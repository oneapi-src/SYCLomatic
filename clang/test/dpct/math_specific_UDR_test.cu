// RUN: dpct --rule-file=%S/../../tools/dpct/DpctOptRules/intel_specific_math.yaml --format-range=none -out-root %T/math_specific_UDR_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/math_specific_UDR_test/math_specific_UDR_test.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>

// CHECK: #include <sycl/ext/intel/math.hpp>


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

// CHECK: void kernelFunc(double *deviceArray) {
// CHECK:   double &d0 = *deviceArray;
// CHECK:   d0 = sycl::ext::intel::math::erfinv(d0);
// CHECK:   d0 = sycl::ext::intel::math::cdfnorm(d0);
// CHECK: }
__global__ void kernelFunc(double *deviceArray) {
  double &d0 = *deviceArray;
  d0 = erfinv(d0);
  d0 = normcdf(d0);
  
}

// CHECK: void kernelFunc(float *deviceArray) {
// CHECK:   float &f0 = *deviceArray;
// CHECK:   f0 = sycl::ext::intel::math::erfinv(f0);
// CHECK:   f0 = sycl::ext::intel::math::cdfnorm(f0);
// CHECK: }
__global__ void kernelFunc(float *deviceArray) {
  float &f0 = *deviceArray;
  f0 = erfinvf(f0);
  f0 = normcdff(f0);
  
}

// CHECK: void testDouble() {
// CHECK:   dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK:   sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK:   const unsigned int NUM = 1;
// CHECK:   const unsigned int bytes = NUM * sizeof(double);
// CHECK:   double *hostArrayDouble = (double *)malloc(bytes);
// CHECK:   memset(hostArrayDouble, 0, bytes);
// CHECK:   double *deviceArrayDouble;
// CHECK:   deviceArrayDouble = (double *)sycl::malloc_device(bytes, q_ct1);
// CHECK:   q_ct1.memcpy(deviceArrayDouble, hostArrayDouble, bytes);
// CHECK:   q_ct1.submit(
// CHECK:    [&](sycl::handler &cgh) {
// CHECK:     dpct::has_capability_or_fail(q_ct1.get_device(), {sycl::aspect::fp64});
// CHECK:     cgh.parallel_for(
// CHECK:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK:       [=](sycl::nd_item<3> item_ct1) {
// CHECK:         kernelFunc(deviceArrayDouble);
// CHECK:       });
// CHECK:   q_ct1.memcpy(hostArrayDouble, deviceArrayDouble, bytes).wait();
// CHECK:   sycl::free(deviceArrayDouble, q_ct1);
// CHECK: }
void testDouble() {
  const unsigned int NUM = 1;
  const unsigned int bytes = NUM * sizeof(double);
  double *hostArrayDouble = (double *)malloc(bytes);
  memset(hostArrayDouble, 0, bytes);
  double *deviceArrayDouble;
  hipMalloc((double **)&deviceArrayDouble, bytes);
  hipMemcpy(deviceArrayDouble, hostArrayDouble, bytes, hipMemcpyHostToDevice);
  kernelFunc<<<1, 1>>>(deviceArrayDouble);
  hipMemcpy(hostArrayDouble, deviceArrayDouble, bytes, hipMemcpyDeviceToHost);
  hipFree(deviceArrayDouble);
}

// CHECK: void testFloat() {
// CHECK:   dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK:   sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK:   const unsigned int NUM = 1;
// CHECK:   const unsigned int bytes = NUM * sizeof(float);
// CHECK:   float *hostArrayFloat = (float *)malloc(bytes);
// CHECK:   memset(hostArrayFloat, 0, bytes);
// CHECK:   float *deviceArrayFloat;
// CHECK:   deviceArrayFloat = (float *)sycl::malloc_device(bytes, q_ct1);
// CHECK:   q_ct1.memcpy(deviceArrayFloat, hostArrayFloat, bytes);
// CHECK:   q_ct1.parallel_for(
// CHECK:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK:       [=](sycl::nd_item<3> item_ct1) {
// CHECK:         kernelFunc(deviceArrayFloat);
// CHECK:       });
// CHECK:   q_ct1.memcpy(hostArrayFloat, deviceArrayFloat, bytes).wait();
// CHECK:   sycl::free(deviceArrayFloat, q_ct1);
// CHECK: }
void testFloat() {
  const unsigned int NUM = 1;
  const unsigned int bytes = NUM * sizeof(float);
  float *hostArrayFloat = (float *)malloc(bytes);
  memset(hostArrayFloat, 0, bytes);
  float *deviceArrayFloat;
  hipMalloc((float **)&deviceArrayFloat, bytes);
  hipMemcpy(deviceArrayFloat, hostArrayFloat, bytes, hipMemcpyHostToDevice);
  kernelFunc<<<1, 1>>>(deviceArrayFloat);
  hipMemcpy(hostArrayFloat, deviceArrayFloat, bytes, hipMemcpyDeviceToHost);
  hipFree(deviceArrayFloat);
}




int main() {
  testDouble();
  testFloat();
}
