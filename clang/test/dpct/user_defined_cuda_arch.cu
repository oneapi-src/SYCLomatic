// RUN: dpct --format-range=none --usm-level=none -out-root %T/user_defined_cuda_arch %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -D__CUDA_ARCH__=200
// RUN: FileCheck --input-file %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.cpp -o %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.o %}
// RUN: dpct --format-range=none --usm-level=none -out-root %T/user_defined_cuda_arch %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -D__CUDA_ARCH__
// RUN: FileCheck --input-file %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.cpp -o %T/user_defined_cuda_arch/user_defined_cuda_arch.dp.o %}

#include <hip/hip_runtime.h>
#include <stdio.h>
//CHECK: #ifdef DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT: void hello(const sycl::stream &[[STREAM:stream_ct1]]) { [[STREAM]] << "foo"; }
#ifdef __CUDA_ARCH__
__device__ void hello() { printf("foo"); }
#else
__device__ void hello() { printf("other"); }
#endif

#if defined(xxx)
__device__ void hello4() { printf("hello2"); }
//CHECK: #elif defined(DPCT_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello4(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#elif defined(__CUDA_ARCH__)
__device__ void hello4() { printf("hello2"); }
#endif

#if defined(xxx)
__device__ void hello5() { printf("hello2"); }
//CHECK: #elif (DPCT_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT: __device__ void hello5() { printf("hello2"); }
#elif (__CUDA_ARCH__ >= 400)
__device__ void hello5() { printf("hello2"); }
#endif

//CHECK: #if defined(DPCT_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello6(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#if defined(__CUDA_ARCH__)
__device__ void hello6() { printf("hello2"); }
#endif

//CHECK: #ifndef DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT: __device__ void hello7() { printf("hello2"); }
//CHECK-NEXT: #else
//CHECK-NEXT: void hello7(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#ifndef __CUDA_ARCH__
__device__ void hello7() { printf("hello2"); }
#else
__device__ void hello7() { printf("hello2"); }
#endif

__device__ void test(){
//CHECK:#if (DPCT_COMPATIBILITY_TEMP >= 400) &&  (DPCT_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT:printf(">400, \n");
//CHECK-NEXT:#elif (DPCT_COMPATIBILITY_TEMP >200)
//CHECK-NEXT:printf(">200, \n");
//CHECK-NEXT:#else
//CHECK-NEXT:[[STREAM]] << "<200, \n";
//CHECK-NEXT:#endif
#if (__CUDA_ARCH__ >= 400) &&  (__CUDA_ARCH__ >= 400)
printf(">400, \n");
#elif (__CUDA_ARCH__ >200)
printf(">200, \n");
#else
printf("<200, \n");
#endif
}

int main(){
    return 0;
}