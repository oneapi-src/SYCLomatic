// RUN: dpct --format-range=none -out-root %T/kernel-usm %s --usm-level=restricted --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/kernel-usm/kernel-usm.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/kernel-usm/kernel-usm.dp.cpp -o %T/kernel-usm/kernel-usm.dp.o %}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <cassert>
#include <vector>

// CHECK: void testDevice(const int *K) {
// CHECK-NEXT: int t = K[0];
// CHECK-NEXT: }
__device__ void testDevice(const int *K) {
  int t = K[0];
}

// CHECK: void testKernelPtr(const int *L, const int *M, int N,
// CHECK-NEXT: const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT: testDevice(L);
// CHECK-NEXT: int gtid = item_ct1.get_group(2) * item_ct1.get_local_range(2) + item_ct1.get_local_id(2);
// CHECK-NEXT: }
__global__ void testKernelPtr(const int *L, const int *M, int N) {
  testDevice(L);
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  dim3 griddim = 2;
  dim3 threaddim = 32;
  int *karg1, *karg2;
  // CHECK: karg1 = sycl::malloc_device<int>(32, q_ct1);
  // CHECK-NEXT: karg2 = sycl::malloc_device<int>(32, q_ct1);
  hipMalloc(&karg1, 32 * sizeof(int));
  hipMalloc(&karg2, 32 * sizeof(int));

  int karg3 = 80;
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class testKernelPtr_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(griddim * threaddim, threaddim),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           testKernelPtr((const int *)karg1, karg2, karg3, item_ct1);
  // CHECK-NEXT:         });
  testKernelPtr<<<griddim, threaddim>>>((const int *)karg1, karg2, karg3);
}

// CHECK:dpct::shared_memory<float, 1> result(32);
// CHECK-NEXT:void my_kernel(float* result, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT: float *resultInGroup) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:  resultInGroup[item_ct1.get_local_id(2)] = item_ct1.get_group(2);
// CHECK-NEXT:  memcpy(&result[item_ct1.get_group(2)*8], resultInGroup, sizeof(float)*8);
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo5 () {
// CHECK-NEXT:  dpct::get_in_order_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::local_accessor<float, 1> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      float * result_ct0 = result.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result[10]);
// CHECK-NEXT:}
__managed__ __device__ float result[32];
__global__ void my_kernel(float* result) {
  __shared__ float resultInGroup[8]; // __shared__ variable
  resultInGroup[threadIdx.x] = blockIdx.x;
  memcpy(&result[blockIdx.x*8], resultInGroup, sizeof(float)*8);
}
int run_foo5 () {
  my_kernel<<<4, 8>>>(result);
  printf("%f ", result[10]);
}

// CHECK:dpct::shared_memory<float, 1> result2(32);
// CHECK-NEXT:int run_foo6 () {
// CHECK-NEXT:  dpct::get_in_order_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::local_accessor<float, 1> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      float * result2_ct0 = result2.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result2_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result2[10]);
// CHECK-NEXT:}
__managed__ float result2[32];
int run_foo6 () {
  my_kernel<<<4, 8>>>(result2);
  printf("%f ", result2[10]);
}

// CHECK:dpct::shared_memory<float, 0> result3;
// CHECK-NEXT:int run_foo7 () {
// CHECK-NEXT:  dpct::get_in_order_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::local_accessor<float, 1> resultInGroup_acc_ct1(sycl::range<1>(8), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      float * result3_ct0 = result3.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel(result3_ct0, item_ct1, resultInGroup_acc_ct1.get_pointer());
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", result3[0]);
// CHECK-NEXT:}
__managed__ float result3;
int run_foo7 () {
  my_kernel<<<4, 8>>>(&result3);
  printf("%f ", result3);
}

// CHECK:dpct::shared_memory<float, 0> in;
// CHECK-NEXT:dpct::shared_memory<float, 0> out;
// CHECK-NEXT:void my_kernel2(float in, float *out, const sycl::nd_item<3> &item_ct1) {
// CHECK-NEXT:  if (item_ct1.get_local_id(2) == 0) {
// CHECK-NEXT:    memcpy(out, &in, sizeof(float));
// CHECK-NEXT:  }
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo8() {
// CHECK-NEXT:  in[0] = 42;
// CHECK-NEXT:  dpct::get_in_order_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      float in_ct0 = in[0];
// CHECK-NEXT:      float * out_ct1 = out.get_ptr();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel2_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 4) * sycl::range<3>(1, 1, 8), sycl::range<3>(1, 1, 8)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          my_kernel2(in_ct0, out_ct1, item_ct1);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  printf("%f ", out[0]);
// CHECK-NEXT:}
__managed__ float in;
__managed__ float out;
__global__ void my_kernel2(float in, float *out) {
  if (threadIdx.x == 0) {
    memcpy(out, &in, sizeof(float));
  }
}
int run_foo8() {
  in = 42;
  my_kernel2<<<4, 8>>>(in, &out);
  printf("%f ", out);
}

struct A{
  int a;
  int* get_pointer(){
    return &a;
  }
};

__global__ void k(int *p){}

// CHECK:int run_foo9() {
// CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.in_order_queue();
// CHECK-NEXT:  std::vector<A> vec(10);
// CHECK-NEXT:  A aa;
// CHECK-NEXT:  q_ct1.submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      int * aa_get_pointer_ct0 = aa.get_pointer();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          k(aa_get_pointer_ct0);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:  q_ct1.submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      int * vec_get_pointer_ct0 = vec[2].get_pointer();
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class k_{{[0-9a-z]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          k(vec_get_pointer_ct0);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
// CHECK-NEXT:}
int run_foo9() {
  std::vector<A> vec(10);
  A aa;
  k<<<1,1>>>(aa.get_pointer());
  k<<<1,1>>>(vec[2].get_pointer());
}

// CHECK:void cuda_pme_forces_dev(float **afn_s) {
// CHECK-NEXT:  // __shared__ variable
// CHECK-NEXT:}
// CHECK-NEXT:int run_foo10() {
// CHECK-NEXT: dpct::get_in_order_queue().submit(
// CHECK-NEXT:   [&](sycl::handler &cgh) {
// CHECK-NEXT:     sycl::local_accessor<float *, 1> afn_s_acc_ct1(sycl::range<1>(3), cgh);
// CHECK-EMPTY:
// CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class cuda_pme_forces_dev_{{[0-9a-z]+}}>>(
// CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:         cuda_pme_forces_dev(afn_s_acc_ct1.get_pointer());
// CHECK-NEXT:       });
// CHECK-NEXT:   });
// CHECK-NEXT:}
__global__ void cuda_pme_forces_dev() {
  __shared__ float *afn_s[3]; // __shared__ variable
}
int run_foo10() {
  cuda_pme_forces_dev<<<1,1>>>();
}

__global__ void my_kernel3(){}
int run_foo11() {
  // CHECK:q_ct1.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  // CHECK-NEXT:q_ct1.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  // CHECK-NEXT:q_ct1.parallel_for<dpct_kernel_name<class my_kernel3_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        my_kernel3();
  // CHECK-NEXT:      });
  my_kernel3<<<1,1,1,hipStreamDefault>>>();
  my_kernel3<<<1,1,1,hipStreamPerThread>>>();
  my_kernel3<<<1,1,1,hipStreamLegacy>>>();
}

int *g_a;

__global__ void foo_kernel3(int *d) {
}
//CHECK:void run_foo(sycl::range<3> c, sycl::range<3> d) {
//CHECK-NEXT:  if (1)
//CHECK-NEXT:      dpct::get_in_order_queue().submit(
//CHECK-NEXT:        [&](sycl::handler &cgh) {
//CHECK-NEXT:          int * g_a_ct0 = &g_a[0];
//CHECK-EMPTY:
//CHECK-NEXT:          cgh.parallel_for<dpct_kernel_name<class foo_kernel3_{{[a-f0-9]+}}>>(
//CHECK-NEXT:            sycl::nd_range<3>(c, sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:            [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:              foo_kernel3(g_a_ct0);
//CHECK-NEXT:            });
//CHECK-NEXT:        });
//CHECK-NEXT:    }
void run_foo(dim3 c, dim3 d) {
  if (1)
    foo_kernel3<<<c, 1>>>(&g_a[0]);
}

__global__ void my_kernel4(int a, int* b, int c, int d, int e, int f, int g){}
int run_foo12() {
  static int aa;
  static int *bb;
  static const int cc = 0;
  static constexpr int dd = 0;

  const int ci = 1;
  int i = 2;

  static const int ee = ci;
  static constexpr int ff = ci;
  static const int gg = i;
  // CHECK:  dpct::get_in_order_queue().submit(
  // CHECK-NEXT:    [&](sycl::handler &cgh) {
  // CHECK-NEXT:      int aa_ct0 = aa;
  // CHECK-NEXT:      int * bb_ct1 = bb;
  // CHECK-NEXT:      int gg_ct6 = gg;
  // CHECK-EMPTY:
  // CHECK-NEXT:      cgh.parallel_for<dpct_kernel_name<class my_kernel4_{{[0-9a-z]+}}>>(
  // CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:          my_kernel4(aa_ct0, bb_ct1, cc, dd, ee, ff, gg_ct6);
  // CHECK-NEXT:        });
  // CHECK-NEXT:    });
  my_kernel4<<<1,1>>>(aa, bb, cc, dd, ee, ff, gg);
}

template<typename T>
__global__ void my_kernel5(T** a_dev){
  __shared__ T* aa;
}

void run_foo13(float* a_host[]) {
  //CHECK:dpct::get_in_order_queue().submit(
  //CHECK-NEXT:  [&](sycl::handler &cgh) {
  //CHECK-NEXT:    sycl::local_accessor<float *, 0> aa_acc_ct1(cgh);
  //CHECK-EMPTY:
  //CHECK-NEXT:    cgh.parallel_for<dpct_kernel_name<class my_kernel5_{{[0-9a-z]+}}, float>>(
  //CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  //CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:        my_kernel5(a_host, aa_acc_ct1);
  //CHECK-NEXT:      });
  //CHECK-NEXT:  });
  my_kernel5<<<1, 1>>>(a_host);
}
