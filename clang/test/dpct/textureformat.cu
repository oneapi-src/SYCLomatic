#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-12.0, cuda-12.1, cuda-12.2, cuda-12.3
// UNSUPPORTED: v12.0, v12.1, v12.2, v12.3
// RUN: dpct --format-range=none --usm-level=none -out-root %T/textureformat %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/textureformat/textureformat.dp.cpp --match-full-lines %s
#include "hip/hip_runtime.h"
__global__ void transformKernel(float* output, hipTextureObject_t texObj, int width, int height, float theta)
{

  unsigned int x = blockIdx.x * blockDim.x + threadIdx.x;
  unsigned int y = blockIdx.y * blockDim.y + threadIdx.y;
  float u = x / (float)width;
  float v = y / (float)height;

  u -= 0.5f;
  v -= 0.5f;
  float tu = u * cosf(theta) - v * sinf(theta) + 0.5f;
  float tv = v * cosf(theta) + u * sinf(theta) + 0.5f;

  output[y * width + x] = tex2D<float>(texObj, tu, tv);
}
// CHECK: dpct::image_wrapper<sycl::float4, 2> tex42;
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1059:{{[0-9]+}}: SYCL only supports 4-channel image format. Adjust the code.
// CHECK-NEXT: */
// CHECK-NEXT: dpct::image_wrapper<sycl::float3, 2> tex32;
static texture<float4, 2> tex42;
static texture<float3, 2> tex32;

int main()
{
  int width = 10;
  int height = 10;
  int size = 10, angle = 10;
  int *h_data;

  // CHECK: /*
  // CHECK-NEXT: DPCT1059:{{[0-9]+}}: SYCL only supports 4-channel image format. Adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::image_channel channelDesc = dpct::image_channel(32, 32, 0, 0, dpct::image_channel_data_type::fp);
  hipChannelFormatDesc channelDesc = hipCreateChannelDesc(32, 32, 0, 0, hipChannelFormatKindFloat);

  // CHECK: /*
  // CHECK-NEXT: DPCT1059:{{[0-9]+}}: SYCL only supports 4-channel image format. Adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::image_channel channelDesc1 = dpct::image_channel::create<sycl::float3>();
  hipChannelFormatDesc channelDesc1 = hipCreateChannelDesc<float3>();

  hipArray* cuArray;
  hipMallocArray(&cuArray, &channelDesc, width, height);

  hipMemcpyToArray(cuArray, 0, 0, h_data, size, hipMemcpyHostToDevice);

  struct hipResourceDesc resDesc;
  memset(&resDesc, 0, sizeof(resDesc));
  resDesc.resType = hipResourceTypeArray;
  resDesc.res.array.array = cuArray;

  struct hipTextureDesc texDesc;
  memset(&texDesc, 0, sizeof(texDesc));
  texDesc.addressMode[0] = hipAddressModeWrap;
  texDesc.addressMode[1] = hipAddressModeWrap;
  texDesc.filterMode = hipFilterModeLinear;
  texDesc.readMode = hipReadModeElementType;
  texDesc.normalizedCoords = 1;

  hipTextureObject_t texObj = 0;
  hipCreateTextureObject(&texObj, &resDesc, &texDesc, NULL);

  float* output;
  hipMalloc(&output, width * height * sizeof(float));

  dim3 dimBlock(16, 16);
  dim3 dimGrid((width + dimBlock.x - 1) / dimBlock.x, (height + dimBlock.y - 1) / dimBlock.y);
  transformKernel<<<dimGrid, dimBlock>>>(output, texObj, width, height, angle);

  hipDestroyTextureObject(texObj);

  hipFreeArray(cuArray);
  hipFree(output);
  return 0;
}

