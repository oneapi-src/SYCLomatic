
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --use-experimental-features=bindless_images -out-root %T/texture/texture_object_bindless_image %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.cpp -o %T/texture/texture_object_bindless_image/texture_object_bindless_image.dp.o %}

// CHECK: void kernel(sycl::ext::oneapi::experimental::sampled_image_handle tex) {
__global__ void kernel(hipTextureObject_t tex) {
  int i;
  float j, k, l, m;
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, (float)i);
  tex1Dfetch<short2>(tex, i);
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, (float)i);
  tex1D<short2>(tex, i);
  // CHECK: i = sycl::ext::oneapi::experimental::read_image<int>(tex, (float)i);
  tex1D(&i, tex, i);
  // CHECK: sycl::ext::oneapi::experimental::read_image<sycl::short2>(tex, sycl::float2(j, k));
  tex2D<short2>(tex, j, k);
  // CHECK: i = sycl::ext::oneapi::experimental::read_image<int>(tex, sycl::float2(j, k));
  tex2D(&i, tex, j, k);
  // CHECK: sycl::ext::oneapi::experimental::read_mipmap<sycl::short2>(tex, j, l);
  tex1DLod<short2>(tex, j, l);
  // CHECK: i = sycl::ext::oneapi::experimental::read_mipmap<int>(tex, j, l);
  tex1DLod(&i, tex, j, l);
  // CHECK: sycl::ext::oneapi::experimental::read_mipmap<sycl::short2>(tex, sycl::float2(j, k), l);
  tex2DLod<short2>(tex, j, k, l);
  // CHECK: i = sycl::ext::oneapi::experimental::read_mipmap<int>(tex, sycl::float2(j, k), l);
  tex2DLod(&i, tex, j, k, l);
  // CHECK: sycl::ext::oneapi::experimental::read_mipmap<sycl::short2>(tex, sycl::float4(j, k, m, 0), l);
  tex3DLod<short2>(tex, j, k, m, l);
  // CHECK: i = sycl::ext::oneapi::experimental::read_mipmap<int>(tex, sycl::float4(j, k, m, 0), l);
  tex3DLod(&i, tex, j, k, m, l);
}

int main() {
  void *input;
  size_t w, h, sizeInBytes, w_offest_src, h_offest_src, w_offest_dest, h_offest_dest;
  unsigned int flag, l;
  hipExtent e;
  // CHECK: dpct::experimental::image_mem_wrapper_ptr pArr, pArr_src;
  hipArray_t pArr, pArr_src;
  // CHECK: dpct::experimental::image_mem_wrapper_ptr pMipMapArr;
  hipMipmappedArray_t pMipMapArr;
  // CHECK: dpct::image_channel desc;
  hipChannelFormatDesc desc;
  // CHECK: pArr = new dpct::experimental::image_mem_wrapper(desc, e);
  hipMalloc3DArray(&pArr, &desc, e);
  // CHECK: pArr = new dpct::experimental::image_mem_wrapper(desc, w, h);
  hipMallocArray(&pArr, &desc, w, h);
  // CHECK: pMipMapArr = new dpct::experimental::image_mem_wrapper(desc, e, sycl::ext::oneapi::experimental::image_type::mipmap, l);
  hipMallocMipmappedArray(&pMipMapArr, &desc, e, l, flag);
  // CHECK: pArr = pMipMapArr->get_mip_level(0);
  hipGetMipmappedArrayLevel(&pArr, pMipMapArr, 0);
  // CHECK: desc = pArr->get_channel();
  // CHECK-NEXT: e = pArr->get_range();
  // CHECK-NEXT: flag = 0;
  hipArrayGetInfo(&desc, &e, &flag, pArr);
  // CHECK: dpct::experimental::dpct_memcpy(pArr_src, w_offest_src, h_offest_src, pArr, w_offest_dest, h_offest_dest, w, h, q_ct1);
  hipMemcpy2DArrayToArray(pArr, w_offest_dest, h_offest_dest, pArr_src,
                           w_offest_src, h_offest_src, w, h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w, w, h, q_ct1);
  hipMemcpy2DFromArray(input, w, pArr, w_offest_src, h_offest_src, w, h,
                        hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w, w, h, q_ct1);
  hipMemcpy2DFromArrayAsync(input, w, pArr, w_offest_src, h_offest_src, w, h,
                             hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w, w, h, q_ct1);
  hipMemcpy2DToArray(pArr, w_offest_dest, h_offest_dest, input, w, w, h,
                      hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w, w, h, q_ct1);
  hipMemcpy2DToArrayAsync(pArr, w_offest_dest, h_offest_dest, input, w, w, h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr_src, w_offest_src, h_offest_src, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  cudaMemcpyArrayToArray(pArr, w_offest_dest, h_offest_dest, pArr_src,
                         w_offest_src, h_offest_src, w * h,
                         hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w * h, q_ct1);
  hipMemcpyFromArray(input, pArr, w_offest_src, h_offest_src, w * h,
                      hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(pArr, w_offest_src, h_offest_src, input, w * h, q_ct1);
  cudaMemcpyFromArrayAsync(input, pArr, w_offest_src, h_offest_src, w * h,
                           hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  hipMemcpyToArray(pArr, w_offest_dest, h_offest_dest, input, w * h,
                    hipMemcpyHostToDevice);
  // CHECK: dpct::experimental::async_dpct_memcpy(input, pArr, w_offest_dest, h_offest_dest, w * h, q_ct1);
  cudaMemcpyToArrayAsync(pArr, w_offest_dest, h_offest_dest, input, w * h,
                         hipMemcpyHostToDevice);

  // CHECK: dpct::image_data resDesc0, resDesc1, resDesc2, resDesc3, resDesc4;
  hipResourceDesc resDesc0, resDesc1, resDesc2, resDesc3, resDesc4;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::matrix);
  resDesc0.resType = hipResourceTypeArray;
  // CHECK: resDesc1.set_data_ptr(pArr);
  resDesc1.res.array.array = pArr;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::matrix);
  resDesc0.resType = hipResourceTypeMipmappedArray;
  // CHECK: resDesc2.set_data_ptr(pMipMapArr);
  resDesc2.res.mipmap.mipmap = pMipMapArr;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::linear);
  resDesc0.resType = hipResourceTypeLinear;
  // CHECK: resDesc3.set_data_ptr(input);
  resDesc3.res.linear.devPtr = input;
  // CHECK: resDesc3.set_channel(desc);
  resDesc3.res.linear.desc = desc;
  // CHECK: resDesc3.set_x(sizeInBytes);
  resDesc3.res.linear.sizeInBytes = sizeInBytes;
  // CHECK: resDesc0.set_data_type(dpct::image_data_type::pitch);
  resDesc0.resType = hipResourceTypePitch2D;
  // CHECK: resDesc4.set_data_ptr(input);
  resDesc4.res.pitch2D.devPtr = input;
  // CHECK: resDesc4.set_channel(desc);
  resDesc4.res.pitch2D.desc = desc;
  // CHECK: resDesc4.set_x(w);
  resDesc4.res.pitch2D.width = w;
  // CHECK: resDesc4.set_y(h);
  resDesc4.res.pitch2D.height = h;
  // CHECK: resDesc4.set_pitch(sizeInBytes);
  resDesc4.res.pitch2D.pitchInBytes = sizeInBytes;
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(pArr);
    resDesc.resType = hipResourceTypeArray;
    resDesc.res.array.array = pArr;
  }
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(pMipMapArr);
    resDesc.resType = hipResourceTypeMipmappedArray;
    resDesc.res.mipmap.mipmap = pMipMapArr;
  }
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(input, sizeInBytes, desc);
    resDesc.resType = hipResourceTypeLinear;
    resDesc.res.linear.devPtr = input;
    resDesc.res.linear.desc = desc;
    resDesc.res.linear.sizeInBytes = sizeInBytes;
  }
  {
    // CHECK: dpct::image_data resDesc;
    hipResourceDesc resDesc;
    // CHECK: resDesc.set_data(input, w, h, sizeInBytes, desc);
    resDesc.resType = hipResourceTypePitch2D;
    resDesc.res.pitch2D.devPtr = input;
    resDesc.res.pitch2D.desc = desc;
    resDesc.res.pitch2D.width = w;
    resDesc.res.pitch2D.height = h;
    resDesc.res.pitch2D.pitchInBytes = sizeInBytes;
  }

  // CHECK: dpct::sampling_info texDesc1, texDesc2, texDesc3, texDesc4;
  hipTextureDesc texDesc1, texDesc2, texDesc3, texDesc4;
  // CHECK: texDesc1.set(sycl::addressing_mode::repeat);
  texDesc1.addressMode[0] = hipAddressModeWrap;
  // CHECK: texDesc2.set(sycl::addressing_mode::clamp_to_edge);
  texDesc2.addressMode[0] = hipAddressModeClamp;
  // CHECK: texDesc3.set(sycl::addressing_mode::mirrored_repeat);
  texDesc3.addressMode[0] = hipAddressModeMirror;
  // CHECK: texDesc4.set(sycl::addressing_mode::clamp);
  texDesc4.addressMode[0] = hipAddressModeBorder;
  // CHECK: texDesc1.set(sycl::filtering_mode::nearest);
  texDesc1.filterMode = hipFilterModePoint;
  // CHECK: texDesc2.set(sycl::filtering_mode::linear);
  texDesc2.filterMode = hipFilterModeLinear;
  // CHECK: texDesc3.set(sycl::coordinate_normalization_mode::unnormalized);
  texDesc3.normalizedCoords = 0;
  // CHECK: texDesc4.set(sycl::coordinate_normalization_mode::normalized);
  texDesc4.normalizedCoords = 1;
  // CHECK: texDesc1.set_max_anisotropy(1);
  texDesc1.maxAnisotropy = 1;
  // CHECK: texDesc1.set_mipmap_filtering(sycl::filtering_mode::nearest);
  texDesc1.mipmapFilterMode = hipFilterModePoint;
  // CHECK: texDesc2.set_mipmap_filtering(sycl::filtering_mode::linear);
  texDesc2.mipmapFilterMode = hipFilterModeLinear;
  // CHECK: texDesc1.set_min_mipmap_level_clamp(1);
  texDesc1.minMipmapLevelClamp = 1;
  // CHECK:  texDesc1.set_max_mipmap_level_clamp(1);
  texDesc1.maxMipmapLevelClamp = 1;

  // CHECK: sycl::ext::oneapi::experimental::sampled_image_handle tex;
  hipTextureObject_t tex;
  // CHECK: tex = dpct::experimental::create_bindless_image(resDesc1, texDesc1);
  hipCreateTextureObject(&tex, &resDesc1, &texDesc1, NULL);
  // CHECK: desc = pArr->get_channel();
  hipGetChannelDesc(&desc, pArr);
  // CHECK: resDesc1 = dpct::experimental::get_data(tex);
  hipGetTextureObjectResourceDesc(&resDesc1, tex);
  // CHECK: texDesc1 = dpct::experimental::get_sampling_info(tex);
  hipGetTextureObjectTextureDesc(&texDesc1, tex);
  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT: sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT: [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:   kernel(tex);
  // CHECK-NEXT: });
  kernel<<<1, 1>>>(tex);
  // CHECK: dpct::experimental::destroy_bindless_image(tex, q_ct1);
  hipDestroyTextureObject(tex);
  // CHECK: delete pArr;
  hipFreeArray(pArr);
  // CHECK: delete pMipMapArr;
  hipFreeMipmappedArray(pMipMapArr);
  return 0;
}
