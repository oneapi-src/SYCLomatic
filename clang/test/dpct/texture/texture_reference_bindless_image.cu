#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-12.0, cuda-12.1, cuda-12.2, cuda-12.3, cuda-12.4, cuda-12.5, cuda-12.6
// UNSUPPORTED: v12.0, v12.1, v12.2, v12.3, v12.4, v12.5, v12.6
// RUN: dpct --format-range=none --use-experimental-features=bindless_images -out-root %T/texture/texture_reference_bindless_image %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture/texture_reference_bindless_image/texture_reference_bindless_image.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/texture/texture_reference_bindless_image/texture_reference_bindless_image.dp.cpp -o %T/texture/texture_reference_bindless_image/texture_reference_bindless_image.dp.o %}

// CHECK: dpct::experimental::bindless_image_wrapper<sycl::short2, 1> tex1;
static texture<short2, 1> tex1;
// CHECK: dpct::experimental::bindless_image_wrapper<sycl::float4, 2> tex2;
static texture<float4, 2> tex2;
// CHECK: dpct::experimental::bindless_image_wrapper<sycl::float4, 3> tex3;
static texture<float4, 3> tex3;

// CHECK: void kernel(sycl::ext::oneapi::experimental::sampled_image_handle tex1,
// CHECK-NEXT:             sycl::ext::oneapi::experimental::sampled_image_handle tex2,
// CHECK-NEXT:             sycl::ext::oneapi::experimental::sampled_image_handle tex3) {
__global__ void kernel() {
  // CHECK: sycl::ext::oneapi::experimental::sample_image<sycl::short2>(tex1, float(1));
  tex1D(tex1, 1);
  // CHECK: sycl::float4 f42 = sycl::ext::oneapi::experimental::sample_image<sycl::float4>(tex2, sycl::float2(1.0f, 1.0f));
  float4 f42 = tex2D(tex2, 1.0f, 1.0f);
  // CHECK: sycl::ext::oneapi::experimental::sample_image<sycl::float4>(tex3, sycl::float3(1.0f, 2.0f, 3.0f));
  tex3D(tex3, 1.0f, 2.0f, 3.0f);
  // CHECK: sycl::ext::oneapi::experimental::sample_mipmap<sycl::short2>(tex1, float(1.0f), 2.0f);
  tex1DLod(tex1, 1.0f, 2.0f);
  // CHECK: sycl::ext::oneapi::experimental::sample_mipmap<sycl::float4>(tex2, sycl::float2(1.0f, 2.0f), 3.0f);
  tex2DLod(tex2, 1.0f, 2.0f, 3.0f);
  // CHECK: sycl::ext::oneapi::experimental::sample_mipmap<sycl::float4>(tex3, sycl::float3(1.0f, 2.0f, 3.0f), 4.0f);
  tex3DLod(tex3, 1.0f, 2.0f, 3.0f, 4.0f);
}

void driverTextureReferenceManagement() {
  // CHECK: sycl::addressing_mode am;
  HIPaddress_mode am;
  // CHECK: dpct::experimental::bindless_image_wrapper_base_p r;
  hipTexRef r;
  int i = 1;
  // CHECK: sycl::filtering_mode fm;
  HIPfilter_mode fm;
  unsigned int u;
  size_t s1 = 1, s2 = 1;
  // CHECK: dpct::device_ptr d;
  hipDeviceptr_t d;
  // CHECK: sycl::ext::oneapi::experimental::image_descriptor D;
  HIP_ARRAY_DESCRIPTOR D;
  // CHECK: dpct::experimental::image_mem_wrapper_ptr a;
  hipArray_t a;
  // CHECK: sycl::image_channel_type f;
  hipArray_Format f;
  // CHECK: am = r->get_addressing_mode();
  hipTexRefGetAddressMode(&am, r, i);
  // CHECK: fm = r->get_filtering_mode();
  hipTexRefGetFilterMode(&fm, r);
  // CHECK: u = r->is_coordinate_normalized() << 1;
  hipTexRefGetFlags(&u, r);
  // CHECK: r->attach(d, s2);
  hipTexRefSetAddress(&s1, r, d, s2);
  // CHECK: r->attach(&D, d, s1);
  hipTexRefSetAddress2D(r, &D, d, s1);
  // CHECK: r->set(am);
  hipTexRefSetAddressMode(r, i, am);
  // CHECK: r->attach(a);
  hipTexRefSetArray(r, a, u);
  // CHECK: r->set(fm);
  hipTexRefSetFilterMode(r, fm);
  // CHECK: /*
  // CHECK-NEXT: DPCT1074:{{.*}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: r->set_coordinate_normalization_mode(u & 0x02);
  hipTexRefSetFlags(r, u);
  // CHECK: r->set_channel_type(f);
  // CHECK-NEXT: r->set_channel_num(i);
  hipTexRefSetFormat(r, f, i);
}

int main() {
  int i;
  // CHECK: tex2.set(sycl::addressing_mode::repeat);
  tex2.addressMode[0] = hipAddressModeWrap;
  // CHECK: auto addressMode = tex2.get_addressing_mode();
  auto addressMode = tex2.addressMode[0];
  // CHECK: tex2.set_channel_size(1, i);
  tex2.channelDesc.x = i;
  // CHECK: i = tex2.get_channel_size();
  i = tex2.channelDesc.x;
  // CHECK: tex2.set_channel_size(2, i);
  tex2.channelDesc.y = i;
  // CHECK: i = tex2.get_channel_size();
  i = tex2.channelDesc.y;
  // CHECK: tex2.set_channel_size(3, i);
  tex2.channelDesc.z = i;
  // CHECK: i = tex2.get_channel_size();
  i = tex2.channelDesc.z;
  // CHECK: tex2.set_channel_size(4, i);
  tex2.channelDesc.w = i;
  // CHECK: i = tex2.get_channel_size();
  i = tex2.channelDesc.w;
  // CHECK: tex2.set_channel_data_type(dpct::image_channel_data_type::fp);
  tex2.channelDesc.f = hipChannelFormatKindFloat;
  // CHECK: auto f = tex2.get_channel_data_type();
  auto f = tex2.channelDesc.f;
  // CHECK: tex2.set_channel(dpct::image_channel::create<sycl::float4>());
  tex2.channelDesc = hipCreateChannelDesc<float4>();
  // CHECK:  auto channelDesc = tex2.get_channel();
  auto channelDesc = tex2.channelDesc;
  // CHECK: tex2.set(sycl::filtering_mode::nearest);
  tex2.filterMode = hipFilterModePoint;
  // CHECK: auto filterMode = tex2.get_filtering_mode();
  auto filterMode = tex2.filterMode;
  // CHECK: tex2.set(sycl::coordinate_normalization_mode::unnormalized);
  tex2.normalized = 0;
  // CHECK: i = tex2.is_coordinate_normalized();
  i = tex2.normalized;
  {
    // CHECK: tex3.set(sycl::coordinate_normalization_mode::normalized);
    tex3.normalized = true;
    // CHECK: tex3.set(sycl::addressing_mode::clamp_to_edge);
    tex3.addressMode[0] = hipAddressModeClamp;
    // CHECK: tex3.set(sycl::addressing_mode::clamp_to_edge);
    tex3.addressMode[1] = hipAddressModeClamp;
    // CHECK: tex3.set(sycl::filtering_mode::linear);
    tex3.filterMode = hipFilterModeLinear;
  }

  void *dataPtr;
  const size_t w = 4;
  const size_t h = 2;
  size_t pitch = sizeof(float4) * 4;
  float4 expect[h * w] = {
      {1, 2, 3, 4},
      {5, 6, 7, 8},
  };
  hipMalloc(&dataPtr, sizeof(expect));
  hipMemcpy(dataPtr, &expect, sizeof(expect), hipMemcpyHostToDevice);
  // CHECK: tex2.attach(dataPtr, pitch * h);
  hipBindTexture(0, tex2, dataPtr, pitch * h);
  // CHECK: tex2.attach(dataPtr, w, h, pitch);
  hipBindTexture2D(0, tex2, dataPtr, w, h, pitch);
  // CHECK: dpct::experimental::image_mem_wrapper_ptr pArr;
  hipArray_t pArr;
  // CHECK: tex2.attach(pArr);
  hipBindTextureToArray(tex2, pArr);
  // CHECK: dpct::experimental::image_mem_wrapper_ptr pMipMapArr;
  hipMipmappedArray_t pMipMapArr;
  // CHECK: tex3.attach(pMipMapArr);
  hipBindTextureToMipmappedArray(tex3, pMipMapArr);
  // CHECK: q_ct1.submit(
  // CHECK-NEXT: [&](sycl::handler &cgh) {
  // CHECK-NEXT:   auto tex1_handle = tex1.get_handle();
  // CHECK-NEXT:   auto tex2_handle = tex2.get_handle();
  // CHECK-NEXT:   auto tex3_handle = tex3.get_handle();
  // CHECK-EMPTY:
  // CHECK-NEXT:   cgh.parallel_for(
  // CHECK-NEXT:     sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:     [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:       kernel(tex1_handle, tex2_handle, tex3_handle);
  // CHECK-NEXT:     });
  // CHECK-NEXT: });
  kernel<<<1, 1>>>();
  // CHECK: tex2.detach();
  hipUnbindTexture(tex2);

  return 0;
}
