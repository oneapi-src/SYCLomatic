// RUN: dpct --format-range=none -out-root %T/Out %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/Out/cudaPointerAttributes.dp.cpp
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>
int main() {
  int N = 2048;
  size_t size = N * sizeof(float);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);

  float *d_A;
  float *d_B;
  float *d_C;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  // CHECK: dpct::pointer_attributes attributes;
  hipPointerAttribute_t attributes;
  // CHECK: dpct::get_pointer_attributes(attributes, h_A);
  hipPointerGetAttributes (&attributes, h_A);
  // CHECK: std::cout << attributes.device << std::endl;
  // CHECK: std::cout << attributes.memory_type << std::endl;
  // CHECK: std::cout << attributes.host_pointer << std::endl;
  // CHECK: std::cout << attributes.device_pointer << std::endl;
  std::cout << attributes.device << std::endl;
  std::cout << attributes.type << std::endl;
  std::cout << attributes.hostPointer << std::endl;
  std::cout << attributes.devicePointer << std::endl;
  // CHECK: dpct::pointer_attributes *attributes2 = new dpct::pointer_attributes();
  hipPointerAttribute_t *attributes2 = new hipPointerAttribute_t();
  // CHECK: dpct::get_pointer_attributes(*attributes2, h_A);
  hipPointerGetAttributes (attributes2, h_A);
  // CHECK: std::cout << attributes2->device << std::endl;
  // CHECK: std::cout << attributes2->memory_type << std::endl;
  std::cout << attributes2->device << std::endl;
  std::cout << attributes2->type << std::endl;
  // CHECK: if (attributes2->memory_type == sycl::usm::alloc::host) {
  // CHECK: } else if (attributes2->memory_type == sycl::usm::alloc::device) {
  // CHECK: } else if (attributes2->memory_type == sycl::usm::alloc::unknown) {
  // CHECK: } else if (attributes2->memory_type == sycl::usm::alloc::unknown) {
  if (attributes2->type == hipMemoryTypeHost) {
    return -1;
  } else if (attributes2->type == hipMemoryTypeDevice) {
    return -1;
  } else if (attributes2->type == hipMemoryTypeManaged) {
    return -1;
  } else if (attributes2->type == cudaMemoryTypeUnregistered) {
    return -1;
  }
}
