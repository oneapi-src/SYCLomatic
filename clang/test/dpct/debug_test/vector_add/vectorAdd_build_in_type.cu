// RUN: dpct --format-range=none --enable-codepin -out-root %T/debug_test/vector_add %s --cuda-include-path="%cuda-path/include" -- -std=c++17  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/vector_add_codepin_sycl/vectorAdd_build_in_type.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/debug_test/vector_add_codepin_sycl/vectorAdd_build_in_type.dp.cpp -o %T/debug_test/vector_add_codepin_sycl/vectorAdd_build_in_type.dp.o %}
//==============================================================
// Copyright 2019 Intel Corporation
//
// SPDX-License-Identifier: MIT
// =============================================================

//CHECK: #include <dpct/codepin/codepin.hpp>
//CHECK: #include "codepin_autogen_util.hpp"
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <stdlib.h>
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float *A, float *B, float *C) {
  A[threadIdx.x] = threadIdx.x + 1.0f;
  B[threadIdx.x] = threadIdx.x + 1.0f;
  C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}
#define CALL(x) if(0!=x){int a=4;}

int main() {
  float *d_A, *d_B, *d_C;
  hipError_t status;

  //CHECK: dpctexp::codepin::get_ptr_size_map()[d_A] = VECTOR_SIZE * sizeof(float);
  CALL(hipMalloc(&d_A, VECTOR_SIZE * sizeof(float)));
  //CHECK: dpctexp::codepin::get_ptr_size_map()[d_B] = VECTOR_SIZE * sizeof(float);
  hipMalloc(&d_B, VECTOR_SIZE * sizeof(float));
  //CHECK: dpctexp::codepin::get_ptr_size_map()[d_C] = VECTOR_SIZE * sizeof(float);
  hipMalloc(&d_C, VECTOR_SIZE * sizeof(float));
  //CHECK: dpctexp::codepin::gen_prolog_API_CP("{{[._0-9a-zA-Z\/\(\)\:]+}}", &q_ct1, "d_A", d_A, "d_B", d_B, "d_C", d_C);
  VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);
  //CHECK: dpctexp::codepin::gen_epilog_API_CP("{{[._0-9a-zA-Z\/\(\)\:]+}}", &q_ct1, "d_A", d_A, "d_B", d_B, "d_C", d_C);
  float Result[VECTOR_SIZE] = {};
 
  status = hipMemcpy(Result, d_C, VECTOR_SIZE * sizeof(float), hipMemcpyDeviceToHost);
  
  if (status != hipSuccess) {
    printf("Could not copy result to host\n");
    exit(EXIT_FAILURE);
  }

  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  for (int i = 0; i < VECTOR_SIZE; i++) {
    if (i % 16 == 0) {
      printf("\n");
    }
    printf("%3.0f ", Result[i]);
  }
  printf("\n");

  return 0;
}
