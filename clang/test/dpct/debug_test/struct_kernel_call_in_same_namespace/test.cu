// RUN: dpct --format-range=none --enable-codepin -out-root %T/debug_test/struct_kernel_call_in_same_namespace %s --cuda-include-path="%cuda-path/include" -- -std=c++17  -x cuda --cuda-host-only
// RUN: FileCheck %S/codepin_autogen_util.hpp.ref --match-full-lines --input-file %T/debug_test/struct_kernel_call_in_same_namespace_codepin_sycl/codepin_autogen_util.hpp
// RUN: FileCheck %S/codepin_autogen_util.hpp.cuda.ref --match-full-lines --input-file %T/debug_test/struct_kernel_call_in_same_namespace_codepin_cuda/codepin_autogen_util.hpp
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/struct_kernel_call_in_same_namespace_codepin_sycl/test.dp.cpp
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/struct_kernel_call_in_same_namespace_codepin_cuda/test.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/debug_test/struct_kernel_call_in_same_namespace_codepin_sycl/test.dp.cpp -o %T/debug_test/struct_kernel_call_in_same_namespace_codepin_sycl/test.dp.o %}
#include <hip/hip_runtime.h>
#include <iostream>
namespace test {
struct P2 {
  int x;
  int y;
};
} // namespace test


struct CCC2 {
  int x;
  int y;
};

namespace test_codepin {
using Point2D = test::P2;
};

//CHECK:  namespace nnn {
namespace nnn {
struct PP2 {
  int x;
  int y;
};

using INT = int;

__global__ void kernel2d(test_codepin::Point2D *a, test_codepin::Point2D *b, test_codepin::Point2D *c) {
  int i = threadIdx.x;
  c[i].x = a[i].x + b[i].x;
  c[i].y = a[i].y + b[i].y;
}

__global__ void kernel2d_org(test::P2 *a, test::P2 *b, test::P2 *c) {
  int i = threadIdx.x;
  c[i].x = a[i].x + b[i].x;
  c[i].y = a[i].y + b[i].y;
}

__global__ void kernel2d_2(PP2 *a, PP2 *b, PP2 *c) {
  int i = threadIdx.x;
  c[i].x = a[i].x + b[i].x;
  c[i].y = a[i].y + b[i].y;
}

__global__ void kerneel_int(INT *a, INT *b, INT *c) {
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

#define NUM 10
void test() {
  test_codepin::Point2D h_2d[NUM];
  for (int i = 0; i < NUM; i++) {
    h_2d[i].x = i;
    h_2d[i].y = i;
  }
  test_codepin::Point2D *d_a2d, *d_b2d, *d_c2d;
  hipMalloc(&d_a2d, sizeof(test_codepin::Point2D) * NUM);
  hipMalloc(&d_b2d, sizeof(test_codepin::Point2D) * NUM);
  hipMalloc(&d_c2d, sizeof(test_codepin::Point2D) * NUM);
  hipMemcpy(d_a2d, h_2d, sizeof(test_codepin::Point2D) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_b2d, h_2d, sizeof(test_codepin::Point2D) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_c2d, h_2d, sizeof(test_codepin::Point2D) * NUM, hipMemcpyHostToDevice);
  kernel2d<<<1, NUM>>>(d_a2d, d_b2d, d_c2d);
  kernel2d_org<<<1, NUM>>>(d_a2d, d_b2d, d_c2d);
  hipDeviceSynchronize();

  PP2 h_pp2_2d[NUM];
  for (int i = 0; i < NUM; i++) {
    h_pp2_2d[i].x = i;
    h_pp2_2d[i].y = i;
  }
  PP2 *d_pp2_a2d, *d_pp2_b2d, *d_pp2_c2d;
  hipMalloc(&d_pp2_a2d, sizeof(PP2) * NUM);
  hipMalloc(&d_pp2_b2d, sizeof(PP2) * NUM);
  hipMalloc(&d_pp2_c2d, sizeof(PP2) * NUM);
  hipMemcpy(d_pp2_a2d, h_pp2_2d, sizeof(PP2) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_pp2_b2d, h_pp2_2d, sizeof(PP2) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_pp2_c2d, h_pp2_2d, sizeof(PP2) * NUM, hipMemcpyHostToDevice);
  kernel2d_2<<<1, NUM>>>(d_pp2_a2d, d_pp2_b2d, d_pp2_c2d);
  hipDeviceSynchronize();

  INT h_int_2d[NUM];
  for (int i = 0; i < NUM; i++) {
    h_int_2d[i] = i;
  }
  INT *d_int_a2d, *d_int_b2d, *d_int_c2d;
  hipMalloc(&d_int_a2d, sizeof(INT) * NUM);
  hipMalloc(&d_int_b2d, sizeof(INT) * NUM);
  hipMalloc(&d_int_c2d, sizeof(INT) * NUM);
  hipMemcpy(d_int_a2d, h_int_2d, sizeof(INT) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_int_b2d, h_int_2d, sizeof(INT) * NUM, hipMemcpyHostToDevice);
  hipMemcpy(d_int_c2d, h_int_2d, sizeof(INT) * NUM, hipMemcpyHostToDevice);
  kerneel_int<<<1, NUM>>>(d_int_a2d, d_int_b2d, d_int_c2d);
  hipDeviceSynchronize();
}
}; // namespace nnn

int main() {

  return 0;
}