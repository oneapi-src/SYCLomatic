// RUN: dpct --format-range=none --enable-codepin -out-root %T/debug_test/vector_add_float3 %s --cuda-include-path="%cuda-path/include" -- -std=c++17  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/vector_add_float3_codepin_sycl/test.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/debug_test/vector_add_float3_codepin_sycl/test.dp.cpp -o %T/debug_test/vector_add_float3_codepin_sycl/test.dp.o %}
//CHECK: #include <dpct/codepin/codepin.hpp>
//CHECK: #include "generated_schema.hpp"

#include <hip/hip_runtime.h>
#include <iostream>
 
// CUDA kernel: Vector addition for float3
__global__
void vectorAdd(float3* a, float3* b, float3* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
    // Check if the thread ID is within the vector size
    if (tid < size) {
        result[tid].x = a[tid].x + b[tid].x;
        result[tid].y = a[tid].y + b[tid].y;
        result[tid].z = a[tid].z + b[tid].z;
    }
}
 
int main() {
    const int vectorSize = 10; // Set the size of the vectors
 
    // Host vectors
    float3 *h_a, *h_b, *h_result;
   
    // Allocate memory for host vectors
    h_a = new float3[vectorSize];
    h_b = new float3[vectorSize];
    h_result = new float3[vectorSize];
 
    // Initialize host vectors
    for (int i = 0; i < vectorSize; ++i) {
        h_a[i] = make_float3(1.0f, 2.0f, 3.0f);
        h_b[i] = make_float3(4.0f, 5.0f, 6.0f);
    }
 
    // Device vectors
    float3 *d_a, *d_b, *d_result;
 
    // Allocate memory for device vectors
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_a)] = vectorSize * sizeof(sycl::float3);
    hipMalloc((void**)&d_a, vectorSize * sizeof(float3));
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_b)] = vectorSize * sizeof(sycl::float3);
    hipMalloc((void**)&d_b, vectorSize * sizeof(float3));
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_result)] = vectorSize * sizeof(sycl::float3);
    hipMalloc((void**)&d_result, vectorSize * sizeof(float3));
 
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, vectorSize * 12, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vectorSize * 12, hipMemcpyHostToDevice);
 
    // Define grid and block dimensions
    dim3 blockDim(256); // 256 threads per block
    dim3 gridDim((vectorSize + blockDim.x - 1) / blockDim.x); // Sufficient blocks to cover the vector size
 
    // Launch the CUDA kernel
    vectorAdd<<<gridDim, blockDim>>>(d_a, d_b, d_result, vectorSize);
 
    // Copy result from device to host
    hipMemcpy(h_result, d_result, vectorSize * 12, hipMemcpyDeviceToHost);
 
    // Print the result
    for (int i = 0; i < 10; ++i) {
        std::cout << "Result[" << i << "]: (" << h_result[i].x << ", " << h_result[i].y << ", " << h_result[i].z << ")\n";
    }
 
    // Free allocated memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_result;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
 
    return 0;
}
