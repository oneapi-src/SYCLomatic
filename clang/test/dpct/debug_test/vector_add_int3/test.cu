// RUN: dpct --format-range=none --enable-codepin -out-root %T/debug_test/vector_add_int3 %s --cuda-include-path="%cuda-path/include" -- -std=c++17  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/vector_add_int3_codepin_sycl/test.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/debug_test/vector_add_int3_codepin_sycl/test.dp.cpp -o %T/debug_test/vector_add_int3_codepin_sycl/test.dp.o %}
//CHECK: #include <dpct/codepin/codepin.hpp>
//CHECK: #include "generated_schema.hpp"

#include <hip/hip_runtime.h>
#include <iostream>
 
// CUDA kernel: Vector addition for int3
__global__
void vectorAdd(int3* a, int3* b, int3* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
    // Check if the thread ID is within the vector size
    if (tid < size) {
        result[tid].x = a[tid].x + b[tid].x;
        result[tid].y = a[tid].y + b[tid].y;
        result[tid].z = a[tid].z + b[tid].z;
    }
}
 
int main() {
    const int vectorSize = 10; // Set the size of the vectors
 
    // Host vectors
    int3 *h_a, *h_b, *h_result;
   
    // Allocate memory for host vectors
    h_a = new int3[vectorSize];
    h_b = new int3[vectorSize];
    h_result = new int3[vectorSize];
 
    // Initialize host vectors
    for (int i = 0; i < vectorSize; ++i) {
        h_a[i] = make_int3(1, 2, 3);
        h_b[i] = make_int3(4, 5, 6);
    }
 
    // Device vectors
    int3 *d_a, *d_b, *d_result;
 
    // Allocate memory for device vectors
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_a)] = vectorSize * sizeof(sycl::int3);
    hipMalloc((void**)&d_a, vectorSize * sizeof(int3));
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_b)] = vectorSize * sizeof(sycl::int3);
    hipMalloc((void**)&d_b, vectorSize * sizeof(int3));
    //CHECK: dpct::experimental::get_ptr_size_map()[*((void**)&d_result)] = vectorSize * sizeof(sycl::int3);
    hipMalloc((void**)&d_result, vectorSize * sizeof(int3));
 
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, vectorSize * 12, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vectorSize * 12, hipMemcpyHostToDevice);
 
    // Define grid and block dimensions
    dim3 blockDim(256); // 256 threads per block
    dim3 gridDim((vectorSize + blockDim.x - 1) / blockDim.x); // Sufficient blocks to cover the vector size
 
    // Launch the CUDA kernel
    //CHECK: dpct::experimental::gen_prolog_API_CP("{{[._0-9a-zA-Z\/\(\)\:]+}}", &q_ct1, "d_a", d_a, "d_b", d_b, "d_result", d_result, "vectorSize", vectorSize);
    vectorAdd<<<gridDim, blockDim, 0, 0>>>(d_a, d_b, d_result, vectorSize);
    //CHECK: dpct::experimental::gen_epilog_API_CP("{{[._0-9a-zA-Z\/\(\)\:]+}}", &q_ct1, "d_a", d_a, "d_b", d_b, "d_result", d_result, "vectorSize", vectorSize);
 
    // Copy result from device to host
    hipMemcpy(h_result, d_result, vectorSize * 12, hipMemcpyDeviceToHost);
 
    // Print the result
    for (int i = 0; i < 10; ++i) {
        std::cout << "Result[" << i << "]: (" << h_result[i].x << ", " << h_result[i].y << ", " << h_result[i].z << ")\n";
    }
 
    // Free allocated memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_result;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
 
    return 0;
}
