// RUN: dpct --format-range=none --enable-codepin -out-root %T/debug_test/vector_add_int3 %s --cuda-include-path="%cuda-path/include" -- -std=c++17  -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/debug_test/vector_add_int3/test.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/debug_test/vector_add_int3/test.dp.cpp -o %T/debug_test/vector_add_int3/test.dp.o %}

#include <hip/hip_runtime.h>
#include <iostream>
 
// CUDA kernel: Vector addition for int3
__global__
void vectorAdd(int3* a, int3* b, int3* result, int size) {
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
 
    // Check if the thread ID is within the vector size
    if (tid < size) {
        result[tid].x = a[tid].x + b[tid].x;
        result[tid].y = a[tid].y + b[tid].y;
        result[tid].z = a[tid].z + b[tid].z;
    }
}
 
int main() {
    const int vectorSize = 10; // Set the size of the vectors
 
    // Host vectors
    int3 *h_a, *h_b, *h_result;
   
    // Allocate memory for host vectors
    h_a = new int3[vectorSize];
    h_b = new int3[vectorSize];
    h_result = new int3[vectorSize];
 
    // Initialize host vectors
    for (int i = 0; i < vectorSize; ++i) {
        h_a[i] = make_int3(1, 2, 3);
        h_b[i] = make_int3(4, 5, 6);
    }
 
    // Device vectors
    int3 *d_a, *d_b, *d_result;
 
    // Allocate memory for device vectors
    hipMalloc((void**)&d_a, vectorSize * sizeof(int3));
    hipMalloc((void**)&d_b, vectorSize * sizeof(int3));
    hipMalloc((void**)&d_result, vectorSize * sizeof(int3));
 
    // Copy host vectors to device
    hipMemcpy(d_a, h_a, vectorSize * 12, hipMemcpyHostToDevice);
    hipMemcpy(d_b, h_b, vectorSize * 12, hipMemcpyHostToDevice);
 
    // Define grid and block dimensions
    dim3 blockDim(256); // 256 threads per block
    dim3 gridDim((vectorSize + blockDim.x - 1) / blockDim.x); // Sufficient blocks to cover the vector size
 
    // Launch the CUDA kernel
    vectorAdd<<<gridDim, blockDim>>>(d_a, d_b, d_result, vectorSize);
 
    // Copy result from device to host
    hipMemcpy(h_result, d_result, vectorSize * 12, hipMemcpyDeviceToHost);
 
    // Print the result
    for (int i = 0; i < 10; ++i) {
        std::cout << "Result[" << i << "]: (" << h_result[i].x << ", " << h_result[i].y << ", " << h_result[i].z << ")\n";
    }
 
    // Free allocated memory
    delete[] h_a;
    delete[] h_b;
    delete[] h_result;
    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
 
    return 0;
}
