// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/bfloat16 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/bfloat16/bfloat16.dp.cpp


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

// CHECK: void foo(sycl::ext::oneapi::bfloat16 *a) {
void foo(__hip_bfloat16 *a) {
  int i = 0;
  float f = 3.0f;
  // CHECK: a[i] = (sycl::ext::oneapi::bfloat16)f;
  a[i] = (__hip_bfloat16)f;
}

// CHECK: void test_conversions_device() {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
__global__ void test_conversions_device() {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);
}

// CHECK: void test_conversions() {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
void test_conversions() {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);
}

int main() { return 0; }
