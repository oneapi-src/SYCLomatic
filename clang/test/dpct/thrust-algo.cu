// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/thrust-algo %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/thrust-algo/thrust-algo.dp.cpp --match-full-lines %s
#include <vector>

#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/reduce.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include <thrust/gather.h>
#include <thrust/binary_search.h>

#include <thrust/find.h>
#include <thrust/sort.h>
#include <thrust/host_vector.h>

void k() {
  std::vector<int> v, v2, v3, v4;

  auto up = [](int x) -> bool { return x < 23; };
  auto bp = [](int x, int y) -> bool { return x < y; };
  auto bo = [](int x, int y) -> int { return x + y; };
  auto gen = []() -> int { return 23; };

  thrust::maximum<int> binary_op;
  thrust::device_vector<int> tv, tv2, tv3, tv4;

  // exclusive_scan

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 0);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 0);
  thrust::exclusive_scan(thrust::host, v.begin(), v.end(), v2.begin());
  thrust::exclusive_scan(thrust::device, tv.begin(), tv.end(), tv2.begin());

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 0);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 0);
  thrust::exclusive_scan(v.begin(), v.end(), v2.begin());
  thrust::exclusive_scan(tv.begin(), tv.end(), tv2.begin());

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 4);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 4);
  thrust::exclusive_scan(thrust::host, v.begin(), v.end(), v2.begin(), 4);
  thrust::exclusive_scan(thrust::device, tv.begin(), tv.end(), tv2.begin(), 4);

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 4);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 4);
  thrust::exclusive_scan(v.begin(), v.end(), v2.begin(), 4);
  thrust::exclusive_scan(tv.begin(), tv.end(), tv2.begin(), 4);

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 1, binary_op);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 1, binary_op);
  thrust::exclusive_scan(thrust::host, v.begin(), v.end(), v2.begin(), 1, binary_op);
  thrust::exclusive_scan(thrust::device, tv.begin(), tv.end(), tv2.begin(), 1, binary_op);

  // CHECK: std::exclusive_scan(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), 1, binary_op);
  // CHECK: std::exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), tv.begin(), tv.end(), tv2.begin(), 1, binary_op);
  thrust::exclusive_scan(v.begin(), v.end(), v2.begin(), 1, binary_op);
  thrust::exclusive_scan(tv.begin(), tv.end(), tv2.begin(), 1, binary_op);

  // exclusive_scan_by_key

  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  thrust::exclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  thrust::exclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin());

  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1);
  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1);
  thrust::exclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1);
  thrust::exclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin(), 1);

  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1, bp);
  // CHECK: oneapi::dpl::exclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1, bp);
  thrust::exclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin(), 1, bp);
  thrust::exclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin(), 1, bp);

  // inclusive_scan_by_key

  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  thrust::inclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin());
  thrust::inclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin());

  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp);
  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp);
  thrust::inclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp);
  thrust::inclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin(), bp);

  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp, bo);
  // CHECK: oneapi::dpl::inclusive_scan_by_segment(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp, bo);
  thrust::inclusive_scan_by_key(thrust::seq, v.begin(), v.end(), v2.begin(), v3.begin(), bp, bo);
  thrust::inclusive_scan_by_key(v.begin(), v.end(), v2.begin(), v3.begin(), bp, bo);

  // partition_point

  // CHECK: dpct::partition_point(oneapi::dpl::execution::seq, v.begin(), v.end(), up);
  thrust::partition_point(thrust::seq, v.begin(), v.end(), up);
  // CHECK: dpct::partition_point(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), up);
  thrust::partition_point(v.begin(), v.end(), up);


  // binary_search

  // CHECK: oneapi::dpl::binary_search(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::binary_search(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  // CHECK: oneapi::dpl::binary_search(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::binary_search(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());

  // CHECK: oneapi::dpl::binary_search(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::binary_search(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  // CHECK: oneapi::dpl::binary_search(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::binary_search(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of thrust::binary_search is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: thrust::binary_search(oneapi::dpl::execution::seq, v.begin(), v.end(), 1);
  thrust::binary_search(thrust::seq, v.begin(), v.end(), 1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of thrust::binary_search is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT:   thrust::binary_search(v.begin(), v.end(), 1);
  thrust::binary_search(v.begin(), v.end(), 1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of thrust::binary_search is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT:   thrust::binary_search(oneapi::dpl::execution::seq, v.begin(), v.end(), 1, bp);
  thrust::binary_search(thrust::seq, v.begin(), v.end(), 1, bp);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of thrust::binary_search is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT:   thrust::binary_search(v.begin(), v.end(), 1, bp);
  thrust::binary_search(v.begin(), v.end(), 1, bp);


  // lower_bound

  // CHECK: oneapi::dpl::lower_bound(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::lower_bound(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  // CHECK: oneapi::dpl::lower_bound(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::lower_bound(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());

  // CHECK: oneapi::dpl::lower_bound(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::lower_bound(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  // CHECK: oneapi::dpl::lower_bound(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::lower_bound(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);


  // upper_bound

  // CHECK: oneapi::dpl::upper_bound(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::upper_bound(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  // CHECK: oneapi::dpl::upper_bound(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::upper_bound(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());

  // CHECK: oneapi::dpl::upper_bound(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::upper_bound(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  // CHECK: oneapi::dpl::upper_bound(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::upper_bound(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);


  // count
  // CHECK: std::count(oneapi::dpl::execution::seq, v.begin(), v.end(), 23);
  thrust::count(thrust::seq, v.begin(), v.end(), 23);
  // CHECK: std::count(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), 23);
  thrust::count(v.begin(), v.end(), 23);


  // generate
  // CHECK: std::generate(oneapi::dpl::execution::seq, v.begin(), v.end(), gen);
  thrust::generate(thrust::seq, v.begin(), v.end(), gen);
  // CHECK: std::generate(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), gen);
  thrust::generate(v.begin(), v.end(), gen);


  // generate_n
  // CHECK: std::generate_n(oneapi::dpl::execution::seq, v.begin(), 23, gen);
  thrust::generate_n(thrust::seq, v.begin(), 23, gen);
  // CHECK: std::generate_n(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), 23, gen);
  thrust::generate_n(v.begin(), 23, gen);


  // merge

  // CHECK: std::merge(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::merge(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  // CHECK: std::merge(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());
  thrust::merge(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin());

  // CHECK: std::merge(oneapi::dpl::execution::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::merge(thrust::seq, v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  // CHECK: std::merge(oneapi::dpl::execution::make_device_policy(q_ct1), v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
  thrust::merge(v.begin(), v.end(), v2.begin(), v2.end(), v3.begin(), bp);
}


void foo(hipStream_t stream) {
  //CHECK:std::vector<int> h;
  //CHECK-NEXT:dpct::device_vector<int> d;
  thrust::host_vector<int> h;
  thrust::device_vector<int> d;

  //thrust::find
  //CHECK:oneapi::dpl::find(oneapi::dpl::execution::seq, h.begin(), h.end(), 1);
  //CHECK-NEXT:oneapi::dpl::find(oneapi::dpl::execution::seq, h.begin(), h.end(), 1);
  //CHECK-NEXT:oneapi::dpl::find(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), 1);
  thrust::find(thrust::seq, h.begin(), h.end(), 1);
  thrust::find(h.begin(), h.end(), 1);
  thrust::find(d.begin(), d.end(), 1);

  //thrust::sort_by_key
  //CHECK:dpct::sort(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), std::greater<int>());
  //CHECK-NEXT:dpct::sort(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin());
  //CHECK-NEXT:dpct::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), h.begin());
  //CHECK-NEXT:dpct::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin());
  //CHECK-NEXT:dpct::sort(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), std::greater<int>());
  //CHECK-NEXT:dpct::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), std::greater<int>());
  thrust::sort_by_key(thrust::seq, h.begin(), h.end(), h.begin(), thrust::greater<int>());
  thrust::sort_by_key(h.begin(), h.end(), h.begin());
  thrust::sort_by_key(d.begin(), d.end(), h.begin());
  thrust::sort_by_key(thrust::device, d.begin(), d.end(), d.begin());
  thrust::sort_by_key(h.begin(), h.end(), h.begin(), thrust::greater<int>());
  thrust::sort_by_key(d.begin(), d.end(), d.begin(), thrust::greater<int>());

  //CHECK:std::multiplies<int> bo1;
  //CHECK-NEXT:std::multiplies<int> bo2;
  thrust::multiplies<int> bo1;
  thrust::multiplies<int> bo2;
  //thrust::inner_product
  //CHECK:dpct::inner_product(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), 1);
  //CHECK-NEXT:dpct::inner_product(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), 1, bo1, bo2);
  //CHECK-NEXT:dpct::inner_product(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), 1);
  //CHECK-NEXT:dpct::inner_product(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), 1);
  //CHECK-NEXT:dpct::inner_product(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), 1, bo1, bo2);
  //CHECK-NEXT:dpct::inner_product(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), 1, bo1, bo2);
  thrust::inner_product(thrust::host, h.begin(), h.end(), h.begin(), 1);
  thrust::inner_product(thrust::device, d.begin(), d.end(), d.begin(), 1, bo1, bo2);
  thrust::inner_product(h.begin(), h.end(), h.begin(), 1);
  thrust::inner_product(d.begin(), d.end(), d.begin(), 1);
  thrust::inner_product(h.begin(), h.end(), h.begin(), 1, bo1, bo2);
  thrust::inner_product(d.begin(), d.end(), d.begin(), 1, bo1, bo2);

  //CHECK:std::not_equal_to<int> bp;
  thrust::not_equal_to<int> bp;
  //thrust::reduce_by_key
  //CHECK:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp, bo1);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp, bo1);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp, bo1);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::seq, h.begin(), h.end(), dpct::constant_iterator<int>(1), h.end(), h.begin());
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp);
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::seq, h.begin(), h.end(), h.begin(), h.end(), h.begin());
  //CHECK-NEXT:oneapi::dpl::reduce_by_segment(oneapi::dpl::execution::make_device_policy(q_ct1), d.begin(), d.end(), d.begin(), d.end(), d.begin());
  thrust::reduce_by_key(thrust::host, h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp, bo1);
  thrust::reduce_by_key(thrust::device, d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp);
  thrust::reduce_by_key(h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp, bo1);
  thrust::reduce_by_key(d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp, bo1);
  thrust::reduce_by_key(thrust::host, h.begin(), h.end(), thrust::constant_iterator<int>(1), h.end(), h.begin());
  thrust::reduce_by_key(h.begin(), h.end(), h.begin(), h.end(), h.begin(), bp);
  thrust::reduce_by_key(d.begin(), d.end(), d.begin(), d.end(), d.begin(), bp);
  thrust::reduce_by_key(h.begin(), h.end(), h.begin(), h.end(), h.begin());
  thrust::reduce_by_key(d.begin(), d.end(), d.begin(), d.end(), d.begin());

  {
    //CHECK:std::vector<int> h_keys,h_values;
    //CHECK-NEXT:dpct::device_vector<int> d_keys, d_values;
    //CHECK-NEXT:oneapi::dpl::equal_to<int> binary_pred;
    thrust::host_vector<int> h_keys,h_values;
    thrust::device_vector<int> d_keys, d_values;
    thrust::equal_to<int> binary_pred;

    //CHECK:dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin());
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin());
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), binary_pred);
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), binary_pred);
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin());
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin());
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
    //CHECK-NEXT:dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
    thrust::unique_by_key(thrust::host, h_keys.begin(), h_keys.end(), h_values.begin());
    thrust::unique_by_key(h_keys.begin(), h_keys.end(), h_values.begin());
    thrust::unique_by_key(thrust::host, h_keys.begin(), h_keys.end(),h_values.begin(), binary_pred);
    thrust::unique_by_key(h_keys.begin(), h_keys.end(),h_values.begin(), binary_pred);
    thrust::unique_by_key(thrust::device, d_keys.begin(), d_keys.end(), d_values.begin());
    thrust::unique_by_key(d_keys.begin(), d_keys.end(), d_values.begin());
    thrust::unique_by_key(thrust::device, d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
    thrust::unique_by_key(d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
  }
}
