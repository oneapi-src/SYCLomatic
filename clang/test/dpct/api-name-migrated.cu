// RUN: dpct --format-range=none --usm-level=none -out-root %T/api-name-migrated %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/api-name-migrated/api-name-migrated.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/api-name-migrated/api-name-migrated.dp.cpp -o %T/api-name-migrated/api-name-migrated.dp.o %}

#include <hip/hip_runtime.h>

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  // CHECK: free(d_A);
  hipHostFree(d_A);

  // CHECK: src = (float *)malloc(bytes);
  hipHostMalloc(&src, bytes, hipHostMallocDefault);

  struct hipPitchedPtr srcGPU;

  // CHECK: sycl::range<3> extent = sycl::range<3>(size * sizeof(float), size, size);
  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  // CHECK: srcGPU = dpct::dpct_malloc(extent);
  hipMalloc3D(&srcGPU, extent);
}

