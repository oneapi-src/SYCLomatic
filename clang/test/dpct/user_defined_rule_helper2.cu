// RUN: dpct --out-root %T/user_defined_rule_helper2 %s --cuda-include-path="%cuda-path/include" --rule-file %S/xpu_2.yaml --format-range=none --usm-level=none
// RUN: FileCheck --input-file %T/user_defined_rule_helper2/user_defined_rule_helper2.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/user_defined_rule_helper2/user_defined_rule_helper2.dp.cpp -o %T/user_defined_rule_helper2/user_defined_rule_helper2.dp.o %}

#ifndef NO_BUILD_TEST

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include "xpu_helper2.h"
#include <hip/hip_runtime.h>

__global__ void foo1_kernel() {}
void foo1() {
  // CHECK: static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream2()).parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)), 
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:     foo1_kernel();
  // CHECK-NEXT:   });
  foo1_kernel<<<1, 1>>>();
}

__global__ void foo2_kernel(double *d) {}

void foo2() {
  double *d;
  // CHECK: d = (double *)dpct::dpct_malloc(sizeof(double));
  // CHECK-NEXT: {
  // CHECK-NEXT:   dpct::has_capability_or_fail(static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream2()).get_device(), {sycl::aspect::fp64});
  // CHECK-EMPTY:
  // CHECK-NEXT:   static_cast<sycl::queue&>(c10::xpu::getCurrentXPUStream2()).submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       auto d_acc_ct0 = dpct::get_access(d, cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)), 
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           foo2_kernel(&d_acc_ct0[0]);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  // CHECK-NEXT: }
  // CHECK-NEXT: dpct::dpct_free(d);
  hipMalloc(&d, sizeof(double));
  foo2_kernel<<<1, 1>>>(d);
  hipFree(d);
}

#endif
