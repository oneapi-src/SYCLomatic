#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/datatypes_test_part2 %s --cuda-include-path="%cuda-path/include" --extra-arg="-std=c++14" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/datatypes_test_part2/datatypes_test_part2.dp.cpp

#include <iostream>
#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <hipfft/hipfft.h>

void case_1(void) {
{
// CHECK: sycl::range<3> var1(1, 1, 1);
// CHECK-NEXT: sycl::range<3> *var2;
// CHECK-NEXT: sycl::range<3> &var3 = var1;
// CHECK-NEXT: sycl::range<3> &&var4 = std::move(var1);
dim3 var1;
dim3 *var2;
dim3 &var3 = var1;
dim3 &&var4 = std::move(var1);
}

{
// CHECK: dpct::err0 var1;
// CHECK-NEXT: dpct::err0 *var2;
// CHECK-NEXT: dpct::err0 &var3 = var1;
// CHECK-NEXT: dpct::err0 &&var4 = std::move(var1);
hipError_t var1;
hipError_t *var2;
hipError_t &var3 = var1;
hipError_t &&var4 = std::move(var1);
}

{
// CHECK: dpct::err0 var1;
// CHECK-NEXT: dpct::err0 *var2;
// CHECK-NEXT: dpct::err0 &var3 = var1;
// CHECK-NEXT: dpct::err0 &&var4 = std::move(var1);
hipError_t var1;
hipError_t *var2;
hipError_t &var3 = var1;
hipError_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hipError_t var1;
hipError_t *var2;
hipError_t &var3 = var1;
hipError_t &&var4 = std::move(var1);
}

{
// CHECK: dpct::event_ptr var1;
// CHECK-NEXT: dpct::event_ptr *var2;
// CHECK-NEXT: dpct::event_ptr &var3 = var1;
// CHECK-NEXT: dpct::event_ptr &&var4 = std::move(var1);
hipEvent_t var1;
hipEvent_t *var2;
hipEvent_t &var3 = var1;
hipEvent_t &&var4 = std::move(var1);
}

{
// CHECK: dpct::blas::descriptor_ptr var1;
// CHECK-NEXT: dpct::blas::descriptor_ptr *var2;
// CHECK-NEXT: dpct::blas::descriptor_ptr &var3 = var1;
// CHECK-NEXT: dpct::blas::descriptor_ptr &&var4 = std::move(var1);
hipblasHandle_t var1;
hipblasHandle_t *var2;
hipblasHandle_t &var3 = var1;
hipblasHandle_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hipblasStatus_t var1;
hipblasStatus_t *var2;
hipblasStatus_t &var3 = var1;
hipblasStatus_t &&var4 = std::move(var1);
}

{
// CHECK: sycl::float2 var1;
// CHECK-NEXT: sycl::float2 *var2;
// CHECK-NEXT: sycl::float2 &var3 = var1;
// CHECK-NEXT: sycl::float2 &&var4 = std::move(var1);
hipComplex var1;
hipComplex *var2;
hipComplex &var3 = var1;
hipComplex &&var4 = std::move(var1);
}

{
// CHECK: sycl::double2 var1;
// CHECK-NEXT: sycl::double2 *var2;
// CHECK-NEXT: sycl::double2 &var3 = var1;
// CHECK-NEXT: sycl::double2 &&var4 = std::move(var1);
hipDoubleComplex var1;
hipDoubleComplex *var2;
hipDoubleComplex &var3 = var1;
hipDoubleComplex &&var4 = std::move(var1);
}

{
// CHECK: oneapi::mkl::uplo var1;
// CHECK-NEXT: oneapi::mkl::uplo *var2;
// CHECK-NEXT: oneapi::mkl::uplo &var3 = var1;
// CHECK-NEXT: oneapi::mkl::uplo &&var4 = std::move(var1);
hipblasFillMode_t var1;
hipblasFillMode_t *var2;
hipblasFillMode_t &var3 = var1;
hipblasFillMode_t &&var4 = std::move(var1);
}

{
// CHECK: oneapi::mkl::diag var1;
// CHECK-NEXT: oneapi::mkl::diag *var2;
// CHECK-NEXT: oneapi::mkl::diag &var3 = var1;
// CHECK-NEXT: oneapi::mkl::diag &&var4 = std::move(var1);
hipblasDiagType_t var1;
hipblasDiagType_t *var2;
hipblasDiagType_t &var3 = var1;
hipblasDiagType_t &&var4 = std::move(var1);
}

{
// CHECK: oneapi::mkl::side var1;
// CHECK-NEXT: oneapi::mkl::side *var2;
// CHECK-NEXT: oneapi::mkl::side &var3 = var1;
// CHECK-NEXT: oneapi::mkl::side &&var4 = std::move(var1);
hipblasSideMode_t var1;
hipblasSideMode_t *var2;
hipblasSideMode_t &var3 = var1;
hipblasSideMode_t &&var4 = std::move(var1);
}

{
// CHECK: oneapi::mkl::transpose var1;
// CHECK-NEXT: oneapi::mkl::transpose *var2;
// CHECK-NEXT: oneapi::mkl::transpose &var3 = var1;
// CHECK-NEXT: oneapi::mkl::transpose &&var4 = std::move(var1);
hipblasOperation_t var1;
hipblasOperation_t *var2;
hipblasOperation_t &var3 = var1;
hipblasOperation_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hipblasStatus_t var1;
hipblasStatus_t *var2;
hipblasStatus_t &var3 = var1;
hipblasStatus_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hipsolverStatus_t var1;
hipsolverStatus_t *var2;
hipsolverStatus_t &var3 = var1;
hipsolverStatus_t &&var4 = std::move(var1);
}

{
// CHECK: int64_t var1;
// CHECK-NEXT: int64_t *var2;
// CHECK-NEXT: int64_t &var3 = var1;
// CHECK-NEXT: int64_t &&var4 = std::move(var1);
hipsolverEigType_t var1;
hipsolverEigType_t *var2;
hipsolverEigType_t &var3 = var1;
hipsolverEigType_t &&var4 = std::move(var1);
}

{
// CHECK: oneapi::mkl::job var1;
// CHECK-NEXT: oneapi::mkl::job *var2;
// CHECK-NEXT: oneapi::mkl::job &var3 = var1;
// CHECK-NEXT: oneapi::mkl::job &&var4 = std::move(var1);
hipsolverEigMode_t var1;
hipsolverEigMode_t *var2;
hipsolverEigMode_t &var3 = var1;
hipsolverEigMode_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hiprandStatus_t var1;
hiprandStatus_t *var2;
hiprandStatus_t &var3 = var1;
hiprandStatus_t &&var4 = std::move(var1);
}

{
// CHECK: int var1;
// CHECK-NEXT: int *var2;
// CHECK-NEXT: int &var3 = var1;
// CHECK-NEXT: int &&var4 = std::move(var1);
hipfftResult_t var1;
hipfftResult_t *var2;
hipfftResult_t &var3 = var1;
hipfftResult_t &&var4 = std::move(var1);
}

{
// CHECK: dpct::queue_ptr var1;
// CHECK-NEXT: dpct::queue_ptr *var2;
// CHECK-NEXT: dpct::queue_ptr &var3 = var1;
// CHECK-NEXT: dpct::queue_ptr &&var4 = std::move(var1);
hipStream_t var1;
hipStream_t *var2;
hipStream_t &var3 = var1;
hipStream_t &&var4 = std::move(var1);
}

{
// CHECK: sycl::queue *var2;
ihipStream_t *var2;
}
}

// case 2
void case_2(void) {
{
// CHECK:  new sycl::range<3>(1, 1, 1);
// CHECK-NEXT:  new sycl::range<3> *();
  new dim3();
  new dim3 *();
}

{
// CHECK:  new dpct::err0();
// CHECK-NEXT:  new dpct::err0 *();
  new hipError_t();
  new hipError_t *();
}

{
// CHECK:  new dpct::err0();
// CHECK-NEXT:  new dpct::err0 *();
  new hipError_t();
  new hipError_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hipError_t();
  new hipError_t *();
}

{
// CHECK:  new dpct::event_ptr();
// CHECK-NEXT:  new dpct::event_ptr *();
  new hipEvent_t();
  new hipEvent_t *();
}

{
// CHECK:  new dpct::blas::descriptor_ptr();
// CHECK-NEXT:  new dpct::blas::descriptor_ptr *();
  new hipblasHandle_t();
  new hipblasHandle_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hipblasStatus_t();
  new hipblasStatus_t *();
}

{
// CHECK:  new sycl::float2();
// CHECK-NEXT:  new sycl::float2 *();
  new hipComplex();
  new hipComplex *();
}

{
// CHECK:  new sycl::double2();
// CHECK-NEXT:  new sycl::double2 *();
  new hipDoubleComplex();
  new hipDoubleComplex *();
}

{
// CHECK:  new oneapi::mkl::uplo();
// CHECK-NEXT:  new oneapi::mkl::uplo *();
  new hipblasFillMode_t();
  new hipblasFillMode_t *();
}

{
// CHECK:  new oneapi::mkl::diag();
// CHECK-NEXT:  new oneapi::mkl::diag *();
  new hipblasDiagType_t();
  new hipblasDiagType_t *();
}

{
// CHECK:  new oneapi::mkl::side();
// CHECK-NEXT:  new oneapi::mkl::side *();
  new hipblasSideMode_t();
  new hipblasSideMode_t *();
}

{
// CHECK:  new oneapi::mkl::transpose();
// CHECK-NEXT:  new oneapi::mkl::transpose *();
  new hipblasOperation_t();
  new hipblasOperation_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hipblasStatus_t();
  new hipblasStatus_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hipsolverStatus_t();
  new hipsolverStatus_t *();
}

{
// CHECK:  new int64_t();
// CHECK-NEXT:  new int64_t *();
  new hipsolverEigType_t();
  new hipsolverEigType_t *();
}

{
// CHECK:  new oneapi::mkl::job();
// CHECK-NEXT:  new oneapi::mkl::job *();
  new hipsolverEigMode_t();
  new hipsolverEigMode_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hiprandStatus_t();
  new hiprandStatus_t *();
}

{
// CHECK:  new int();
// CHECK-NEXT:  new int *();
  new hipfftResult_t();
  new hipfftResult_t *();
}

{
// CHECK:  new dpct::queue_ptr();
// CHECK-NEXT:  new dpct::queue_ptr *();
  new hipStream_t();
  new hipStream_t *();
}

{
// CHECK: new sycl::queue *();
  new ihipStream_t *();
}
}

// case 3
// CHECK: sycl::range<3> foo0();
// CHECK-NEXT: sycl::range<3> *foo1();
// CHECK-NEXT: sycl::range<3> &foo2();
dim3 foo0();
dim3 *foo1();
dim3 &foo2();

// CHECK: dpct::err0 foo3();
// CHECK-NEXT: dpct::err0 *foo4();
// CHECK-NEXT: dpct::err0 &foo5();
hipError_t foo3();
hipError_t *foo4();
hipError_t &foo5();

// CHECK: dpct::err0 foo6();
// CHECK-NEXT: dpct::err0 *foo7();
// CHECK-NEXT: dpct::err0 &foo8();
hipError_t foo6();
hipError_t *foo7();
hipError_t &foo8();

// CHECK: int foo9();
// CHECK-NEXT: int *foo10();
// CHECK-NEXT: int &foo11();
hipError_t foo9();
hipError_t *foo10();
hipError_t &foo11();

// CHECK: dpct::event_ptr foo12();
// CHECK-NEXT: dpct::event_ptr *foo13();
// CHECK-NEXT: dpct::event_ptr &foo14();
hipEvent_t foo12();
hipEvent_t *foo13();
hipEvent_t &foo14();

// CHECK: dpct::blas::descriptor_ptr foo15();
// CHECK-NEXT: dpct::blas::descriptor_ptr *foo16();
// CHECK-NEXT: dpct::blas::descriptor_ptr &foo17();
hipblasHandle_t foo15();
hipblasHandle_t *foo16();
hipblasHandle_t &foo17();

// CHECK: int foo18();
// CHECK-NEXT: int *foo19();
// CHECK-NEXT: int &foo20();
hipblasStatus_t foo18();
hipblasStatus_t *foo19();
hipblasStatus_t &foo20();

// CHECK: sycl::float2 foo21();
// CHECK-NEXT: sycl::float2 *foo22();
// CHECK-NEXT: sycl::float2 &foo23();
hipComplex foo21();
hipComplex *foo22();
hipComplex &foo23();

// CHECK: sycl::double2 foo24();
// CHECK-NEXT: sycl::double2 *foo25();
// CHECK-NEXT: sycl::double2 &foo26();
hipDoubleComplex foo24();
hipDoubleComplex *foo25();
hipDoubleComplex &foo26();

// CHECK: oneapi::mkl::uplo foo27();
// CHECK-NEXT: oneapi::mkl::uplo *foo28();
// CHECK-NEXT: oneapi::mkl::uplo &foo29();
hipblasFillMode_t foo27();
hipblasFillMode_t *foo28();
hipblasFillMode_t &foo29();

// CHECK: oneapi::mkl::diag foo30();
// CHECK-NEXT: oneapi::mkl::diag *foo31();
// CHECK-NEXT: oneapi::mkl::diag &foo32();
hipblasDiagType_t foo30();
hipblasDiagType_t *foo31();
hipblasDiagType_t &foo32();

// CHECK: oneapi::mkl::side foo33();
// CHECK-NEXT: oneapi::mkl::side *foo34();
// CHECK-NEXT: oneapi::mkl::side &foo35();
hipblasSideMode_t foo33();
hipblasSideMode_t *foo34();
hipblasSideMode_t &foo35();

// CHECK: oneapi::mkl::transpose foo36();
// CHECK-NEXT: oneapi::mkl::transpose *foo37();
// CHECK-NEXT: oneapi::mkl::transpose &foo38();
hipblasOperation_t foo36();
hipblasOperation_t *foo37();
hipblasOperation_t &foo38();

// CHECK: int foo39();
// CHECK-NEXT: int *foo40();
// CHECK-NEXT: int &foo41();
hipblasStatus_t foo39();
hipblasStatus_t *foo40();
hipblasStatus_t &foo41();

// CHECK: int foo42();
// CHECK-NEXT: int *foo43();
// CHECK-NEXT: int &foo44();
hipsolverStatus_t foo42();
hipsolverStatus_t *foo43();
hipsolverStatus_t &foo44();

// CHECK: int64_t foo45();
// CHECK-NEXT: int64_t *foo46();
// CHECK-NEXT: int64_t &foo47();
hipsolverEigType_t foo45();
hipsolverEigType_t *foo46();
hipsolverEigType_t &foo47();

// CHECK: oneapi::mkl::job foo48();
// CHECK-NEXT: oneapi::mkl::job *foo49();
// CHECK-NEXT: oneapi::mkl::job &foo50();
hipsolverEigMode_t foo48();
hipsolverEigMode_t *foo49();
hipsolverEigMode_t &foo50();

// CHECK: int foo51();
// CHECK-NEXT: int *foo52();
// CHECK-NEXT: int &foo53();
hiprandStatus_t foo51();
hiprandStatus_t *foo52();
hiprandStatus_t &foo53();

// CHECK: int foo54();
// CHECK-NEXT: int *foo55();
// CHECK-NEXT: int &foo56();
hipfftResult_t foo54();
hipfftResult_t *foo55();
hipfftResult_t &foo56();

// CHECK: dpct::queue_ptr foo57();
// CHECK-NEXT: dpct::queue_ptr *foo58();
// CHECK-NEXT: dpct::queue_ptr &foo59();
hipStream_t foo57();
hipStream_t *foo58();
hipStream_t &foo59();

// CHECK: sycl::queue foo_1();
// CHECK-NEXT: sycl::queue *foo_2();
ihipStream_t foo_1();
ihipStream_t *foo_2();


// case 4
template <typename T> struct S {};

// CHECK: template <> struct S<sycl::range<3>> {};
// CHECK-NEXT: template <> struct S<sycl::range<3> *> {};
// CHECK-NEXT: template <> struct S<sycl::range<3> &> {};
// CHECK-NEXT: template <> struct S<sycl::range<3> &&> {};
template <> struct S<dim3> {};
template <> struct S<dim3 *> {};
template <> struct S<dim3 &> {};
template <> struct S<dim3 &&> {};

// CHECK: template <> struct S<dpct::err0> {};
// CHECK-NEXT: template <> struct S<dpct::err0 *> {};
// CHECK-NEXT: template <> struct S<dpct::err0 &> {};
// CHECK-NEXT: template <> struct S<dpct::err0 &&> {};
template <> struct S<hipError_t> {};
template <> struct S<hipError_t *> {};
template <> struct S<hipError_t &> {};
template <> struct S<hipError_t &&> {};

// CHECK: template <> struct S<int> {};
// CHECK-NEXT: template <> struct S<int *> {};
// CHECK-NEXT: template <> struct S<int &> {};
// CHECK-NEXT: template <> struct S<int &&> {};
template <> struct S<hipError_t> {};
template <> struct S<hipError_t *> {};
template <> struct S<hipError_t &> {};
template <> struct S<hipError_t &&> {};

// CHECK: template <> struct S<dpct::event_ptr> {};
// CHECK-NEXT: template <> struct S<dpct::event_ptr *> {};
// CHECK-NEXT: template <> struct S<dpct::event_ptr &> {};
// CHECK-NEXT: template <> struct S<dpct::event_ptr &&> {};
template <> struct S<hipEvent_t> {};
template <> struct S<hipEvent_t *> {};
template <> struct S<hipEvent_t &> {};
template <> struct S<hipEvent_t &&> {};

// CHECK: template <> struct S<dpct::blas::descriptor_ptr> {};
// CHECK-NEXT: template <> struct S<dpct::blas::descriptor_ptr *> {};
// CHECK-NEXT: template <> struct S<dpct::blas::descriptor_ptr &> {};
// CHECK-NEXT: template <> struct S<dpct::blas::descriptor_ptr &&> {};
template <> struct S<hipblasHandle_t> {};
template <> struct S<hipblasHandle_t *> {};
template <> struct S<hipblasHandle_t &> {};
template <> struct S<hipblasHandle_t &&> {};


// CHECK: template <> struct S<sycl::float2> {};
// CHECK-NEXT: template <> struct S<sycl::float2 *> {};
// CHECK-NEXT: template <> struct S<sycl::float2 &> {};
// CHECK-NEXT: template <> struct S<sycl::float2 &&> {};
template <> struct S<hipComplex> {};
template <> struct S<hipComplex *> {};
template <> struct S<hipComplex &> {};
template <> struct S<hipComplex &&> {};

// CHECK: template <> struct S<sycl::double2> {};
// CHECK-NEXT: template <> struct S<sycl::double2 *> {};
// CHECK-NEXT: template <> struct S<sycl::double2 &> {};
// CHECK-NEXT: template <> struct S<sycl::double2 &&> {};
template <> struct S<hipDoubleComplex> {};
template <> struct S<hipDoubleComplex *> {};
template <> struct S<hipDoubleComplex &> {};
template <> struct S<hipDoubleComplex &&> {};

// CHECK: template <> struct S<oneapi::mkl::uplo> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::uplo *> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::uplo &> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::uplo &&> {};
template <> struct S<hipblasFillMode_t> {};
template <> struct S<hipblasFillMode_t *> {};
template <> struct S<hipblasFillMode_t &> {};
template <> struct S<hipblasFillMode_t &&> {};

// CHECK: template <> struct S<oneapi::mkl::diag> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::diag *> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::diag &> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::diag &&> {};
template <> struct S<hipblasDiagType_t> {};
template <> struct S<hipblasDiagType_t *> {};
template <> struct S<hipblasDiagType_t &> {};
template <> struct S<hipblasDiagType_t &&> {};

// CHECK: template <> struct S<oneapi::mkl::side> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::side *> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::side &> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::side &&> {};
template <> struct S<hipblasSideMode_t> {};
template <> struct S<hipblasSideMode_t *> {};
template <> struct S<hipblasSideMode_t &> {};
template <> struct S<hipblasSideMode_t &&> {};

// CHECK: template <> struct S<oneapi::mkl::transpose> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::transpose *> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::transpose &> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::transpose &&> {};
template <> struct S<hipblasOperation_t> {};
template <> struct S<hipblasOperation_t *> {};
template <> struct S<hipblasOperation_t &> {};
template <> struct S<hipblasOperation_t &&> {};

// CHECK: template <> struct S<int> {};
// CHECK-NEXT: template <> struct S<int *> {};
// CHECK-NEXT: template <> struct S<int &> {};
// CHECK-NEXT: template <> struct S<int &&> {};
template <> struct S<hipblasStatus_t> {};
template <> struct S<hipblasStatus_t *> {};
template <> struct S<hipblasStatus_t &> {};
template <> struct S<hipblasStatus_t &&> {};

// CHECK: template <> struct S<int> {};
// CHECK-NEXT: template <> struct S<int *> {};
// CHECK-NEXT: template <> struct S<int &> {};
// CHECK-NEXT: template <> struct S<int &&> {};
template <> struct S<hipsolverStatus_t> {};
template <> struct S<hipsolverStatus_t *> {};
template <> struct S<hipsolverStatus_t &> {};
template <> struct S<hipsolverStatus_t &&> {};

// CHECK: template <> struct S<int64_t> {};
// CHECK-NEXT: template <> struct S<int64_t *> {};
// CHECK-NEXT: template <> struct S<int64_t &> {};
// CHECK-NEXT: template <> struct S<int64_t &&> {};
template <> struct S<hipsolverEigType_t> {};
template <> struct S<hipsolverEigType_t *> {};
template <> struct S<hipsolverEigType_t &> {};
template <> struct S<hipsolverEigType_t &&> {};

// CHECK: template <> struct S<oneapi::mkl::job> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::job *> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::job &> {};
// CHECK-NEXT: template <> struct S<oneapi::mkl::job &&> {};
template <> struct S<hipsolverEigMode_t> {};
template <> struct S<hipsolverEigMode_t *> {};
template <> struct S<hipsolverEigMode_t &> {};
template <> struct S<hipsolverEigMode_t &&> {};

// CHECK: template <> struct S<int> {};
// CHECK-NEXT: template <> struct S<int *> {};
// CHECK-NEXT: template <> struct S<int &> {};
// CHECK-NEXT: template <> struct S<int &&> {};
template <> struct S<hiprandStatus_t> {};
template <> struct S<hiprandStatus_t *> {};
template <> struct S<hiprandStatus_t &> {};
template <> struct S<hiprandStatus_t &&> {};

// CHECK: template <> struct S<int> {};
// CHECK-NEXT: template <> struct S<int *> {};
// CHECK-NEXT: template <> struct S<int &> {};
// CHECK-NEXT: template <> struct S<int &&> {};
template <> struct S<hipfftResult_t> {};
template <> struct S<hipfftResult_t *> {};
template <> struct S<hipfftResult_t &> {};
template <> struct S<hipfftResult_t &&> {};

// CHECK: template <> struct S<dpct::queue_ptr> {};
// CHECK-NEXT: template <> struct S<dpct::queue_ptr *> {};
// CHECK-NEXT: template <> struct S<dpct::queue_ptr &> {};
// CHECK-NEXT: template <> struct S<dpct::queue_ptr &&> {};
template <> struct S<hipStream_t> {};
template <> struct S<hipStream_t *> {};
template <> struct S<hipStream_t &> {};
template <> struct S<hipStream_t &&> {};


// case 5
template <typename T> void template_foo() {}
void case_5(){

// CHECK: template_foo<sycl::range<3>>();
// CHECK-NEXT: template_foo<sycl::range<3> *>();
// CHECK-NEXT: template_foo<sycl::range<3> &>();
// CHECK-NEXT: template_foo<sycl::range<3> &&>();
template_foo<dim3>();
template_foo<dim3 *>();
template_foo<dim3 &>();
template_foo<dim3 &&>();

// CHECK: template_foo<dpct::err0>();
// CHECK-NEXT: template_foo<dpct::err0 *>();
// CHECK-NEXT: template_foo<dpct::err0 &>();
// CHECK-NEXT: template_foo<dpct::err0 &&>();
template_foo<hipError_t>();
template_foo<hipError_t *>();
template_foo<hipError_t &>();
template_foo<hipError_t &&>();

// CHECK: template_foo<dpct::err0>();
// CHECK-NEXT: template_foo<dpct::err0 *>();
// CHECK-NEXT: template_foo<dpct::err0 &>();
// CHECK-NEXT: template_foo<dpct::err0 &&>();
template_foo<hipError_t>();
template_foo<hipError_t *>();
template_foo<hipError_t &>();
template_foo<hipError_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hipError_t>();
template_foo<hipError_t *>();
template_foo<hipError_t &>();
template_foo<hipError_t &&>();

// CHECK: template_foo<dpct::event_ptr>();
// CHECK-NEXT: template_foo<dpct::event_ptr *>();
// CHECK-NEXT: template_foo<dpct::event_ptr &>();
// CHECK-NEXT: template_foo<dpct::event_ptr &&>();
template_foo<hipEvent_t>();
template_foo<hipEvent_t *>();
template_foo<hipEvent_t &>();
template_foo<hipEvent_t &&>();

// CHECK: template_foo<dpct::blas::descriptor_ptr>();
// CHECK-NEXT: template_foo<dpct::blas::descriptor_ptr *>();
// CHECK-NEXT: template_foo<dpct::blas::descriptor_ptr &>();
// CHECK-NEXT: template_foo<dpct::blas::descriptor_ptr &&>();
template_foo<hipblasHandle_t>();
template_foo<hipblasHandle_t *>();
template_foo<hipblasHandle_t &>();
template_foo<hipblasHandle_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hipblasStatus_t>();
template_foo<hipblasStatus_t *>();
template_foo<hipblasStatus_t &>();
template_foo<hipblasStatus_t &&>();

// CHECK: template_foo<sycl::float2>();
// CHECK-NEXT: template_foo<sycl::float2 *>();
// CHECK-NEXT: template_foo<sycl::float2 &>();
// CHECK-NEXT: template_foo<sycl::float2 &&>();
template_foo<hipComplex>();
template_foo<hipComplex *>();
template_foo<hipComplex &>();
template_foo<hipComplex &&>();

// CHECK: template_foo<sycl::double2>();
// CHECK-NEXT: template_foo<sycl::double2 *>();
// CHECK-NEXT: template_foo<sycl::double2 &>();
// CHECK-NEXT: template_foo<sycl::double2 &&>();
template_foo<hipDoubleComplex>();
template_foo<hipDoubleComplex *>();
template_foo<hipDoubleComplex &>();
template_foo<hipDoubleComplex &&>();

// CHECK: template_foo<oneapi::mkl::uplo>();
// CHECK-NEXT: template_foo<oneapi::mkl::uplo *>();
// CHECK-NEXT: template_foo<oneapi::mkl::uplo &>();
// CHECK-NEXT: template_foo<oneapi::mkl::uplo &&>();
template_foo<hipblasFillMode_t>();
template_foo<hipblasFillMode_t *>();
template_foo<hipblasFillMode_t &>();
template_foo<hipblasFillMode_t &&>();

// CHECK: template_foo<oneapi::mkl::diag>();
// CHECK-NEXT: template_foo<oneapi::mkl::diag *>();
// CHECK-NEXT: template_foo<oneapi::mkl::diag &>();
// CHECK-NEXT: template_foo<oneapi::mkl::diag &&>();
template_foo<hipblasDiagType_t>();
template_foo<hipblasDiagType_t *>();
template_foo<hipblasDiagType_t &>();
template_foo<hipblasDiagType_t &&>();

// CHECK: template_foo<oneapi::mkl::side>();
// CHECK-NEXT: template_foo<oneapi::mkl::side *>();
// CHECK-NEXT: template_foo<oneapi::mkl::side &>();
// CHECK-NEXT: template_foo<oneapi::mkl::side &&>();
template_foo<hipblasSideMode_t>();
template_foo<hipblasSideMode_t *>();
template_foo<hipblasSideMode_t &>();
template_foo<hipblasSideMode_t &&>();

// CHECK: template_foo<oneapi::mkl::transpose>();
// CHECK-NEXT: template_foo<oneapi::mkl::transpose *>();
// CHECK-NEXT: template_foo<oneapi::mkl::transpose &>();
// CHECK-NEXT: template_foo<oneapi::mkl::transpose &&>();
template_foo<hipblasOperation_t>();
template_foo<hipblasOperation_t *>();
template_foo<hipblasOperation_t &>();
template_foo<hipblasOperation_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hipblasStatus_t>();
template_foo<hipblasStatus_t *>();
template_foo<hipblasStatus_t &>();
template_foo<hipblasStatus_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hipsolverStatus_t>();
template_foo<hipsolverStatus_t *>();
template_foo<hipsolverStatus_t &>();
template_foo<hipsolverStatus_t &&>();

// CHECK: template_foo<int64_t>();
// CHECK-NEXT: template_foo<int64_t *>();
// CHECK-NEXT: template_foo<int64_t &>();
// CHECK-NEXT: template_foo<int64_t &&>();
template_foo<hipsolverEigType_t>();
template_foo<hipsolverEigType_t *>();
template_foo<hipsolverEigType_t &>();
template_foo<hipsolverEigType_t &&>();

// CHECK: template_foo<oneapi::mkl::job>();
// CHECK-NEXT: template_foo<oneapi::mkl::job *>();
// CHECK-NEXT: template_foo<oneapi::mkl::job &>();
// CHECK-NEXT: template_foo<oneapi::mkl::job &&>();
template_foo<hipsolverEigMode_t>();
template_foo<hipsolverEigMode_t *>();
template_foo<hipsolverEigMode_t &>();
template_foo<hipsolverEigMode_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hiprandStatus_t>();
template_foo<hiprandStatus_t *>();
template_foo<hiprandStatus_t &>();
template_foo<hiprandStatus_t &&>();

// CHECK: template_foo<int>();
// CHECK-NEXT: template_foo<int *>();
// CHECK-NEXT: template_foo<int &>();
// CHECK-NEXT: template_foo<int &&>();
template_foo<hipfftResult_t>();
template_foo<hipfftResult_t *>();
template_foo<hipfftResult_t &>();
template_foo<hipfftResult_t &&>();

// CHECK: template_foo<dpct::queue_ptr>();
// CHECK-NEXT: template_foo<dpct::queue_ptr *>();
// CHECK-NEXT: template_foo<dpct::queue_ptr &>();
// CHECK-NEXT: template_foo<dpct::queue_ptr &&>();
template_foo<hipStream_t>();
template_foo<hipStream_t *>();
template_foo<hipStream_t &>();
template_foo<hipStream_t &&>();

// CHECK: template_foo<sycl::queue>();
// CHECK-NEXT: template_foo<sycl::queue *>();
// CHECK-NEXT: template_foo<sycl::queue &>();
// CHECK-NEXT: template_foo<sycl::queue &&>();
template_foo<ihipStream_t>();
template_foo<ihipStream_t *>();
template_foo<ihipStream_t &>();
template_foo<ihipStream_t &&>();

}


// case 6
// CHECK: using UT0 = sycl::range<3>;
// CHECK-NEXT: using UT1 = sycl::range<3> *;
// CHECK-NEXT: using UT2 = sycl::range<3> &;
// CHECK-NEXT: using UT3 = sycl::range<3> &&;
using UT0 = dim3;
using UT1 = dim3 *;
using UT2 = dim3 &;
using UT3 = dim3 &&;

// CHECK: using UT4 = dpct::err0;
// CHECK-NEXT: using UT5 = dpct::err0 *;
// CHECK-NEXT: using UT6 = dpct::err0 &;
// CHECK-NEXT: using UT7 = dpct::err0 &&;
using UT4 = hipError_t;
using UT5 = hipError_t *;
using UT6 = hipError_t &;
using UT7 = hipError_t &&;

// CHECK: using UT8 = dpct::err0;
// CHECK-NEXT: using UT9 = dpct::err0 *;
// CHECK-NEXT: using UT10 = dpct::err0 &;
// CHECK-NEXT: using UT11 = dpct::err0 &&;
using UT8 = hipError_t;
using UT9 = hipError_t *;
using UT10 = hipError_t &;
using UT11 = hipError_t &&;

// CHECK: using UT12 = int;
// CHECK-NEXT: using UT13 = int *;
// CHECK-NEXT: using UT14 = int &;
// CHECK-NEXT: using UT15 = int &&;
using UT12 = hipError_t;
using UT13 = hipError_t *;
using UT14 = hipError_t &;
using UT15 = hipError_t &&;

// CHECK: using UT16 = dpct::event_ptr;
// CHECK-NEXT: using UT17 = dpct::event_ptr *;
// CHECK-NEXT: using UT18 = dpct::event_ptr &;
// CHECK-NEXT: using UT19 = dpct::event_ptr &&;
using UT16 = hipEvent_t;
using UT17 = hipEvent_t *;
using UT18 = hipEvent_t &;
using UT19 = hipEvent_t &&;

// CHECK: using UT20 = dpct::blas::descriptor_ptr;
// CHECK-NEXT: using UT21 = dpct::blas::descriptor_ptr *;
// CHECK-NEXT: using UT22 = dpct::blas::descriptor_ptr &;
// CHECK-NEXT: using UT23 = dpct::blas::descriptor_ptr &&;
using UT20 = hipblasHandle_t;
using UT21 = hipblasHandle_t *;
using UT22 = hipblasHandle_t &;
using UT23 = hipblasHandle_t &&;

// CHECK: using UT24 = int;
// CHECK-NEXT: using UT25 = int *;
// CHECK-NEXT: using UT26 = int &;
// CHECK-NEXT: using UT27 = int &&;
using UT24 = hipblasStatus_t;
using UT25 = hipblasStatus_t *;
using UT26 = hipblasStatus_t &;
using UT27 = hipblasStatus_t &&;

// CHECK: using UT28 = sycl::float2;
// CHECK-NEXT: using UT29 = sycl::float2 *;
// CHECK-NEXT: using UT30 = sycl::float2 &;
// CHECK-NEXT: using UT31 = sycl::float2 &&;
using UT28 = hipComplex;
using UT29 = hipComplex *;
using UT30 = hipComplex &;
using UT31 = hipComplex &&;

// CHECK: using UT32 = sycl::double2;
// CHECK-NEXT: using UT33 = sycl::double2 *;
// CHECK-NEXT: using UT34 = sycl::double2 &;
// CHECK-NEXT: using UT35 = sycl::double2 &&;
using UT32 = hipDoubleComplex;
using UT33 = hipDoubleComplex *;
using UT34 = hipDoubleComplex &;
using UT35 = hipDoubleComplex &&;

// CHECK: using UT36 = oneapi::mkl::uplo;
// CHECK-NEXT: using UT37 = oneapi::mkl::uplo *;
// CHECK-NEXT: using UT38 = oneapi::mkl::uplo &;
// CHECK-NEXT: using UT39 = oneapi::mkl::uplo &&;
using UT36 = hipblasFillMode_t;
using UT37 = hipblasFillMode_t *;
using UT38 = hipblasFillMode_t &;
using UT39 = hipblasFillMode_t &&;

// CHECK: using UT40 = oneapi::mkl::diag;
// CHECK-NEXT: using UT41 = oneapi::mkl::diag *;
// CHECK-NEXT: using UT42 = oneapi::mkl::diag &;
// CHECK-NEXT: using UT43 = oneapi::mkl::diag &&;
using UT40 = hipblasDiagType_t;
using UT41 = hipblasDiagType_t *;
using UT42 = hipblasDiagType_t &;
using UT43 = hipblasDiagType_t &&;

// CHECK: using UT44 = oneapi::mkl::side;
// CHECK-NEXT: using UT45 = oneapi::mkl::side *;
// CHECK-NEXT: using UT46 = oneapi::mkl::side &;
// CHECK-NEXT: using UT47 = oneapi::mkl::side &&;
using UT44 = hipblasSideMode_t;
using UT45 = hipblasSideMode_t *;
using UT46 = hipblasSideMode_t &;
using UT47 = hipblasSideMode_t &&;

// CHECK: using UT48 = oneapi::mkl::transpose;
// CHECK-NEXT: using UT49 = oneapi::mkl::transpose *;
// CHECK-NEXT: using UT50 = oneapi::mkl::transpose &;
// CHECK-NEXT: using UT51 = oneapi::mkl::transpose &&;
using UT48 = hipblasOperation_t;
using UT49 = hipblasOperation_t *;
using UT50 = hipblasOperation_t &;
using UT51 = hipblasOperation_t &&;

// CHECK: using UT52 = int;
// CHECK-NEXT: using UT53 = int *;
// CHECK-NEXT: using UT54 = int &;
// CHECK-NEXT: using UT55 = int &&;
using UT52 = hipblasStatus_t;
using UT53 = hipblasStatus_t *;
using UT54 = hipblasStatus_t &;
using UT55 = hipblasStatus_t &&;

// CHECK: using UT56 = int;
// CHECK-NEXT: using UT57 = int *;
// CHECK-NEXT: using UT58 = int &;
// CHECK-NEXT: using UT59 = int &&;
using UT56 = hipsolverStatus_t;
using UT57 = hipsolverStatus_t *;
using UT58 = hipsolverStatus_t &;
using UT59 = hipsolverStatus_t &&;

// CHECK: using UT60 = int64_t;
// CHECK-NEXT: using UT61 = int64_t *;
// CHECK-NEXT: using UT62 = int64_t &;
// CHECK-NEXT: using UT63 = int64_t &&;
using UT60 = hipsolverEigType_t;
using UT61 = hipsolverEigType_t *;
using UT62 = hipsolverEigType_t &;
using UT63 = hipsolverEigType_t &&;

// CHECK: using UT64 = oneapi::mkl::job;
// CHECK-NEXT: using UT65 = oneapi::mkl::job *;
// CHECK-NEXT: using UT66 = oneapi::mkl::job &;
// CHECK-NEXT: using UT67 = oneapi::mkl::job &&;
using UT64 = hipsolverEigMode_t;
using UT65 = hipsolverEigMode_t *;
using UT66 = hipsolverEigMode_t &;
using UT67 = hipsolverEigMode_t &&;

// CHECK: using UT68 = int;
// CHECK-NEXT: using UT69 = int *;
// CHECK-NEXT: using UT70 = int &;
// CHECK-NEXT: using UT71 = int &&;
using UT68 = hiprandStatus_t;
using UT69 = hiprandStatus_t *;
using UT70 = hiprandStatus_t &;
using UT71 = hiprandStatus_t &&;

// CHECK: using UT72 = int;
// CHECK-NEXT: using UT73 = int *;
// CHECK-NEXT: using UT74 = int &;
// CHECK-NEXT: using UT75 = int &&;
using UT72 = hipfftResult_t;
using UT73 = hipfftResult_t *;
using UT74 = hipfftResult_t &;
using UT75 = hipfftResult_t &&;

// CHECK: using UT76 = dpct::queue_ptr;
// CHECK-NEXT: using UT77 = dpct::queue_ptr *;
// CHECK-NEXT: using UT78 = dpct::queue_ptr &;
// CHECK-NEXT: using UT79 = dpct::queue_ptr &&;
using UT76 = hipStream_t;
using UT77 = hipStream_t *;
using UT78 = hipStream_t &;
using UT79 = hipStream_t &&;

// CHECK: using UT_1 = sycl::queue;
// CHECK-NEXT: using UT_2 = sycl::queue *;
// CHECK-NEXT: using UT_3 = sycl::queue &;
// CHECK-NEXT: using UT_4 = sycl::queue &&;
using UT_1 = ihipStream_t;
using UT_2 = ihipStream_t *;
using UT_3 = ihipStream_t &;
using UT_4 = ihipStream_t &&;



// case 7
// CHECK: typedef sycl::range<3> T0;
// CHECK-NEXT: typedef sycl::range<3>* T1;
// CHECK-NEXT: typedef sycl::range<3>& T2;
// CHECK-NEXT: typedef sycl::range<3>&& T3;
typedef dim3 T0;
typedef dim3* T1;
typedef dim3& T2;
typedef dim3&& T3;

// CHECK: typedef dpct::err0 T4;
// CHECK-NEXT: typedef dpct::err0* T5;
// CHECK-NEXT: typedef dpct::err0& T6;
// CHECK-NEXT: typedef dpct::err0&& T7;
typedef hipError_t T4;
typedef hipError_t* T5;
typedef hipError_t& T6;
typedef hipError_t&& T7;

// CHECK: typedef dpct::err0 T8;
// CHECK-NEXT: typedef dpct::err0* T9;
// CHECK-NEXT: typedef dpct::err0& T10;
// CHECK-NEXT: typedef dpct::err0&& T11;
typedef hipError_t T8;
typedef hipError_t* T9;
typedef hipError_t& T10;
typedef hipError_t&& T11;

// CHECK: typedef int T12;
// CHECK-NEXT: typedef int* T13;
// CHECK-NEXT: typedef int& T14;
// CHECK-NEXT: typedef int&& T15;
typedef hipError_t T12;
typedef hipError_t* T13;
typedef hipError_t& T14;
typedef hipError_t&& T15;

// CHECK: typedef dpct::event_ptr T16;
// CHECK-NEXT: typedef dpct::event_ptr* T17;
// CHECK-NEXT: typedef dpct::event_ptr& T18;
// CHECK-NEXT: typedef dpct::event_ptr&& T19;
typedef hipEvent_t T16;
typedef hipEvent_t* T17;
typedef hipEvent_t& T18;
typedef hipEvent_t&& T19;

// CHECK: typedef dpct::blas::descriptor_ptr T20;
// CHECK-NEXT: typedef dpct::blas::descriptor_ptr* T21;
// CHECK-NEXT: typedef dpct::blas::descriptor_ptr& T22;
// CHECK-NEXT: typedef dpct::blas::descriptor_ptr&& T23;
typedef hipblasHandle_t T20;
typedef hipblasHandle_t* T21;
typedef hipblasHandle_t& T22;
typedef hipblasHandle_t&& T23;

// CHECK: typedef int T24;
// CHECK-NEXT: typedef int* T25;
// CHECK-NEXT: typedef int& T26;
// CHECK-NEXT: typedef int&& T27;
typedef hipblasStatus_t T24;
typedef hipblasStatus_t* T25;
typedef hipblasStatus_t& T26;
typedef hipblasStatus_t&& T27;

// CHECK: typedef sycl::float2 T28;
// CHECK-NEXT: typedef sycl::float2* T29;
// CHECK-NEXT: typedef sycl::float2& T30;
// CHECK-NEXT: typedef sycl::float2&& T31;
typedef hipComplex T28;
typedef hipComplex* T29;
typedef hipComplex& T30;
typedef hipComplex&& T31;

// CHECK: typedef sycl::double2 T32;
// CHECK-NEXT: typedef sycl::double2* T33;
// CHECK-NEXT: typedef sycl::double2& T34;
// CHECK-NEXT: typedef sycl::double2&& T35;
typedef hipDoubleComplex T32;
typedef hipDoubleComplex* T33;
typedef hipDoubleComplex& T34;
typedef hipDoubleComplex&& T35;

// CHECK: typedef oneapi::mkl::uplo T36;
// CHECK-NEXT: typedef oneapi::mkl::uplo* T37;
// CHECK-NEXT: typedef oneapi::mkl::uplo& T38;
// CHECK-NEXT: typedef oneapi::mkl::uplo&& T39;
typedef hipblasFillMode_t T36;
typedef hipblasFillMode_t* T37;
typedef hipblasFillMode_t& T38;
typedef hipblasFillMode_t&& T39;

// CHECK: typedef oneapi::mkl::diag T40;
// CHECK-NEXT: typedef oneapi::mkl::diag* T41;
// CHECK-NEXT: typedef oneapi::mkl::diag& T42;
// CHECK-NEXT: typedef oneapi::mkl::diag&& T43;
typedef hipblasDiagType_t T40;
typedef hipblasDiagType_t* T41;
typedef hipblasDiagType_t& T42;
typedef hipblasDiagType_t&& T43;

// CHECK: typedef oneapi::mkl::side T44;
// CHECK-NEXT: typedef oneapi::mkl::side* T45;
// CHECK-NEXT: typedef oneapi::mkl::side& T46;
// CHECK-NEXT: typedef oneapi::mkl::side&& T47;
typedef hipblasSideMode_t T44;
typedef hipblasSideMode_t* T45;
typedef hipblasSideMode_t& T46;
typedef hipblasSideMode_t&& T47;

// CHECK: typedef oneapi::mkl::transpose T48;
// CHECK-NEXT: typedef oneapi::mkl::transpose* T49;
// CHECK-NEXT: typedef oneapi::mkl::transpose& T50;
// CHECK-NEXT: typedef oneapi::mkl::transpose&& T51;
typedef hipblasOperation_t T48;
typedef hipblasOperation_t* T49;
typedef hipblasOperation_t& T50;
typedef hipblasOperation_t&& T51;

// CHECK: typedef int T52;
// CHECK-NEXT: typedef int* T53;
// CHECK-NEXT: typedef int& T54;
// CHECK-NEXT: typedef int&& T55;
typedef hipblasStatus_t T52;
typedef hipblasStatus_t* T53;
typedef hipblasStatus_t& T54;
typedef hipblasStatus_t&& T55;

// CHECK: typedef int T56;
// CHECK-NEXT: typedef int* T57;
// CHECK-NEXT: typedef int& T58;
// CHECK-NEXT: typedef int&& T59;
typedef hipsolverStatus_t T56;
typedef hipsolverStatus_t* T57;
typedef hipsolverStatus_t& T58;
typedef hipsolverStatus_t&& T59;

// CHECK: typedef int64_t T60;
// CHECK-NEXT: typedef int64_t* T61;
// CHECK-NEXT: typedef int64_t& T62;
// CHECK-NEXT: typedef int64_t&& T63;
typedef hipsolverEigType_t T60;
typedef hipsolverEigType_t* T61;
typedef hipsolverEigType_t& T62;
typedef hipsolverEigType_t&& T63;

// CHECK: typedef oneapi::mkl::job T64;
// CHECK-NEXT: typedef oneapi::mkl::job* T65;
// CHECK-NEXT: typedef oneapi::mkl::job& T66;
// CHECK-NEXT: typedef oneapi::mkl::job&& T67;
typedef hipsolverEigMode_t T64;
typedef hipsolverEigMode_t* T65;
typedef hipsolverEigMode_t& T66;
typedef hipsolverEigMode_t&& T67;

// CHECK: typedef int T68;
// CHECK-NEXT: typedef int* T69;
// CHECK-NEXT: typedef int& T70;
// CHECK-NEXT: typedef int&& T71;
typedef hiprandStatus_t T68;
typedef hiprandStatus_t* T69;
typedef hiprandStatus_t& T70;
typedef hiprandStatus_t&& T71;

// CHECK: typedef int T72;
// CHECK-NEXT: typedef int* T73;
// CHECK-NEXT: typedef int& T74;
// CHECK-NEXT: typedef int&& T75;
typedef hipfftResult_t T72;
typedef hipfftResult_t* T73;
typedef hipfftResult_t& T74;
typedef hipfftResult_t&& T75;

// CHECK: typedef dpct::queue_ptr T76;
// CHECK-NEXT: typedef dpct::queue_ptr* T77;
// CHECK-NEXT: typedef dpct::queue_ptr& T78;
// CHECK-NEXT: typedef dpct::queue_ptr&& T79;
typedef hipStream_t T76;
typedef hipStream_t* T77;
typedef hipStream_t& T78;
typedef hipStream_t&& T79;

// CHECK: typedef sycl::queue T_1;
// CHECK-NEXT: typedef sycl::queue* T_2;
// CHECK-NEXT: typedef sycl::queue& T_3;
// CHECK-NEXT: typedef sycl::queue&& T_4;
typedef ihipStream_t T_1;
typedef ihipStream_t* T_2;
typedef ihipStream_t& T_3;
typedef ihipStream_t&& T_4;


// case 8
__device__ void foo_t(){

{
// CHECK: #define T8_0 sycl::range<3>
// CHECK-NEXT: #define T8_1 sycl::range<3> *
// CHECK-NEXT: #define T8_2 sycl::range<3> &
// CHECK-NEXT: #define T8_3 sycl::range<3> &&
// CHECK-NEXT:     T8_0 a1(1, 1, 1);
// CHECK-NEXT:     T8_1 a2;
// CHECK-NEXT:     T8_2 a3=a1;
// CHECK-NEXT:     T8_3 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_0 dim3
#define T8_1 dim3 *
#define T8_2 dim3 &
#define T8_3 dim3 &&
    T8_0 a1;
    T8_1 a2;
    T8_2 a3=a1;
    T8_3 a4=std::move(a1);
}

{
// CHECK: #define T8_4 dpct::err0
// CHECK-NEXT: #define T8_5 dpct::err0 *
// CHECK-NEXT: #define T8_6 dpct::err0 &
// CHECK-NEXT: #define T8_7 dpct::err0 &&
// CHECK-NEXT:     T8_4 a1;
// CHECK-NEXT:     T8_5 a2;
// CHECK-NEXT:     T8_6 a3=a1;
// CHECK-NEXT:     T8_7 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_4 hipError_t
#define T8_5 hipError_t *
#define T8_6 hipError_t &
#define T8_7 hipError_t &&
    T8_4 a1;
    T8_5 a2;
    T8_6 a3=a1;
    T8_7 a4=std::move(a1);
}

{
// CHECK: #define T8_8 dpct::err0
// CHECK-NEXT: #define T8_9 dpct::err0 *
// CHECK-NEXT: #define T8_10 dpct::err0 &
// CHECK-NEXT: #define T8_11 dpct::err0 &&
// CHECK-NEXT:     T8_8 a1;
// CHECK-NEXT:     T8_9 a2;
// CHECK-NEXT:     T8_10 a3=a1;
// CHECK-NEXT:     T8_11 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_8 hipError_t
#define T8_9 hipError_t *
#define T8_10 hipError_t &
#define T8_11 hipError_t &&
    T8_8 a1;
    T8_9 a2;
    T8_10 a3=a1;
    T8_11 a4=std::move(a1);
}

{
// CHECK: #define T8_12 int
// CHECK-NEXT: #define T8_13 int *
// CHECK-NEXT: #define T8_14 int &
// CHECK-NEXT: #define T8_15 int &&
// CHECK-NEXT:     T8_12 a1;
// CHECK-NEXT:     T8_13 a2;
// CHECK-NEXT:     T8_14 a3=a1;
// CHECK-NEXT:     T8_15 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_12 hipError_t
#define T8_13 hipError_t *
#define T8_14 hipError_t &
#define T8_15 hipError_t &&
    T8_12 a1;
    T8_13 a2;
    T8_14 a3=a1;
    T8_15 a4=std::move(a1);
}

{
// CHECK: #define T8_16 dpct::event_ptr
// CHECK-NEXT: #define T8_17 dpct::event_ptr *
// CHECK-NEXT: #define T8_18 dpct::event_ptr &
// CHECK-NEXT: #define T8_19 dpct::event_ptr &&
// CHECK-NEXT:     T8_16 a1;
// CHECK-NEXT:     T8_17 a2;
// CHECK-NEXT:     T8_18 a3=a1;
// CHECK-NEXT:     T8_19 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_16 hipEvent_t
#define T8_17 hipEvent_t *
#define T8_18 hipEvent_t &
#define T8_19 hipEvent_t &&
    T8_16 a1;
    T8_17 a2;
    T8_18 a3=a1;
    T8_19 a4=std::move(a1);
}

{
// CHECK: /*
// CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
// CHECK-NEXT: */
// CHECK-NEXT: #define T8_20 hipblasHandle_t
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
// CHECK-NEXT: */
// CHECK-NEXT: #define T8_21 hipblasHandle_t *
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
// CHECK-NEXT: */
// CHECK-NEXT: #define T8_22 hipblasHandle_t &
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1021:{{[0-9]+}}: Migration of hipblasHandle_t in __global__ or __device__ function is not supported. You may need to redesign the code.
// CHECK-NEXT: */
// CHECK-NEXT: #define T8_23 hipblasHandle_t &&
// CHECK-NEXT:     T8_20 a1;
// CHECK-NEXT:     T8_21 a2;
// CHECK-NEXT:     T8_22 a3=a1;
// CHECK-NEXT:     T8_23 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_20 hipblasHandle_t
#define T8_21 hipblasHandle_t *
#define T8_22 hipblasHandle_t &
#define T8_23 hipblasHandle_t &&
    T8_20 a1;
    T8_21 a2;
    T8_22 a3=a1;
    T8_23 a4=std::move(a1);
}

{
// CHECK: #define T8_24 int
// CHECK-NEXT: #define T8_25 int *
// CHECK-NEXT: #define T8_26 int &
// CHECK-NEXT: #define T8_27 int &&
// CHECK-NEXT:     T8_24 a1;
// CHECK-NEXT:     T8_25 a2;
// CHECK-NEXT:     T8_26 a3=a1;
// CHECK-NEXT:     T8_27 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_24 hipblasStatus_t
#define T8_25 hipblasStatus_t *
#define T8_26 hipblasStatus_t &
#define T8_27 hipblasStatus_t &&
    T8_24 a1;
    T8_25 a2;
    T8_26 a3=a1;
    T8_27 a4=std::move(a1);
}

{
// CHECK: #define T8_28 sycl::float2
// CHECK-NEXT: #define T8_29 sycl::float2 *
// CHECK-NEXT: #define T8_30 sycl::float2 &
// CHECK-NEXT: #define T8_31 sycl::float2 &&
// CHECK-NEXT:     T8_28 a1;
// CHECK-NEXT:     T8_29 a2;
// CHECK-NEXT:     T8_30 a3=a1;
// CHECK-NEXT:     T8_31 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_28 hipComplex
#define T8_29 hipComplex *
#define T8_30 hipComplex &
#define T8_31 hipComplex &&
    T8_28 a1;
    T8_29 a2;
    T8_30 a3=a1;
    T8_31 a4=std::move(a1);
}

{
// CHECK: #define T8_32 sycl::double2
// CHECK-NEXT: #define T8_33 sycl::double2 *
// CHECK-NEXT: #define T8_34 sycl::double2 &
// CHECK-NEXT: #define T8_35 sycl::double2 &&
// CHECK-NEXT:     T8_32 a1;
// CHECK-NEXT:     T8_33 a2;
// CHECK-NEXT:     T8_34 a3=a1;
// CHECK-NEXT:     T8_35 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_32 hipDoubleComplex
#define T8_33 hipDoubleComplex *
#define T8_34 hipDoubleComplex &
#define T8_35 hipDoubleComplex &&
    T8_32 a1;
    T8_33 a2;
    T8_34 a3=a1;
    T8_35 a4=std::move(a1);
}

{
// CHECK: #define T8_36 oneapi::mkl::uplo
// CHECK-NEXT: #define T8_37 oneapi::mkl::uplo *
// CHECK-NEXT: #define T8_38 oneapi::mkl::uplo &
// CHECK-NEXT: #define T8_39 oneapi::mkl::uplo &&
// CHECK-NEXT:     T8_36 a1;
// CHECK-NEXT:     T8_37 a2;
// CHECK-NEXT:     T8_38 a3=a1;
// CHECK-NEXT:     T8_39 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_36 hipblasFillMode_t
#define T8_37 hipblasFillMode_t *
#define T8_38 hipblasFillMode_t &
#define T8_39 hipblasFillMode_t &&
    T8_36 a1;
    T8_37 a2;
    T8_38 a3=a1;
    T8_39 a4=std::move(a1);
}

{
// CHECK: #define T8_40 oneapi::mkl::diag
// CHECK-NEXT: #define T8_41 oneapi::mkl::diag *
// CHECK-NEXT: #define T8_42 oneapi::mkl::diag &
// CHECK-NEXT: #define T8_43 oneapi::mkl::diag &&
// CHECK-NEXT:     T8_40 a1;
// CHECK-NEXT:     T8_41 a2;
// CHECK-NEXT:     T8_42 a3=a1;
// CHECK-NEXT:     T8_43 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_40 hipblasDiagType_t
#define T8_41 hipblasDiagType_t *
#define T8_42 hipblasDiagType_t &
#define T8_43 hipblasDiagType_t &&
    T8_40 a1;
    T8_41 a2;
    T8_42 a3=a1;
    T8_43 a4=std::move(a1);
}

{
// CHECK: #define T8_44 oneapi::mkl::side
// CHECK-NEXT: #define T8_45 oneapi::mkl::side *
// CHECK-NEXT: #define T8_46 oneapi::mkl::side &
// CHECK-NEXT: #define T8_47 oneapi::mkl::side &&
// CHECK-NEXT:     T8_44 a1;
// CHECK-NEXT:     T8_45 a2;
// CHECK-NEXT:     T8_46 a3=a1;
// CHECK-NEXT:     T8_47 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_44 hipblasSideMode_t
#define T8_45 hipblasSideMode_t *
#define T8_46 hipblasSideMode_t &
#define T8_47 hipblasSideMode_t &&
    T8_44 a1;
    T8_45 a2;
    T8_46 a3=a1;
    T8_47 a4=std::move(a1);
}

{
// CHECK: #define T8_48 oneapi::mkl::transpose
// CHECK-NEXT: #define T8_49 oneapi::mkl::transpose *
// CHECK-NEXT: #define T8_50 oneapi::mkl::transpose &
// CHECK-NEXT: #define T8_51 oneapi::mkl::transpose &&
// CHECK-NEXT:     T8_48 a1;
// CHECK-NEXT:     T8_49 a2;
// CHECK-NEXT:     T8_50 a3=a1;
// CHECK-NEXT:     T8_51 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_48 hipblasOperation_t
#define T8_49 hipblasOperation_t *
#define T8_50 hipblasOperation_t &
#define T8_51 hipblasOperation_t &&
    T8_48 a1;
    T8_49 a2;
    T8_50 a3=a1;
    T8_51 a4=std::move(a1);
}

{
// CHECK: #define T8_52 int
// CHECK-NEXT: #define T8_53 int *
// CHECK-NEXT: #define T8_54 int &
// CHECK-NEXT: #define T8_55 int &&
// CHECK-NEXT:     T8_52 a1;
// CHECK-NEXT:     T8_53 a2;
// CHECK-NEXT:     T8_54 a3=a1;
// CHECK-NEXT:     T8_55 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_52 hipblasStatus_t
#define T8_53 hipblasStatus_t *
#define T8_54 hipblasStatus_t &
#define T8_55 hipblasStatus_t &&
    T8_52 a1;
    T8_53 a2;
    T8_54 a3=a1;
    T8_55 a4=std::move(a1);
}

{
// CHECK: #define T8_56 int
// CHECK-NEXT: #define T8_57 int *
// CHECK-NEXT: #define T8_58 int &
// CHECK-NEXT: #define T8_59 int &&
// CHECK-NEXT:     T8_56 a1;
// CHECK-NEXT:     T8_57 a2;
// CHECK-NEXT:     T8_58 a3=a1;
// CHECK-NEXT:     T8_59 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_56 hipsolverStatus_t
#define T8_57 hipsolverStatus_t *
#define T8_58 hipsolverStatus_t &
#define T8_59 hipsolverStatus_t &&
    T8_56 a1;
    T8_57 a2;
    T8_58 a3=a1;
    T8_59 a4=std::move(a1);
}

{
// CHECK: #define T8_60 int64_t
// CHECK-NEXT: #define T8_61 int64_t *
// CHECK-NEXT: #define T8_62 int64_t &
// CHECK-NEXT: #define T8_63 int64_t &&
// CHECK-NEXT:     T8_60 a1;
// CHECK-NEXT:     T8_61 a2;
// CHECK-NEXT:     T8_62 a3=a1;
// CHECK-NEXT:     T8_63 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_60 hipsolverEigType_t
#define T8_61 hipsolverEigType_t *
#define T8_62 hipsolverEigType_t &
#define T8_63 hipsolverEigType_t &&
    T8_60 a1;
    T8_61 a2;
    T8_62 a3=a1;
    T8_63 a4=std::move(a1);
}

{
// CHECK: #define T8_64 oneapi::mkl::job
// CHECK-NEXT: #define T8_65 oneapi::mkl::job *
// CHECK-NEXT: #define T8_66 oneapi::mkl::job &
// CHECK-NEXT: #define T8_67 oneapi::mkl::job &&
// CHECK-NEXT:     T8_64 a1;
// CHECK-NEXT:     T8_65 a2;
// CHECK-NEXT:     T8_66 a3=a1;
// CHECK-NEXT:     T8_67 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_64 hipsolverEigMode_t
#define T8_65 hipsolverEigMode_t *
#define T8_66 hipsolverEigMode_t &
#define T8_67 hipsolverEigMode_t &&
    T8_64 a1;
    T8_65 a2;
    T8_66 a3=a1;
    T8_67 a4=std::move(a1);
}

{
// CHECK: #define T8_68 int
// CHECK-NEXT: #define T8_69 int *
// CHECK-NEXT: #define T8_70 int &
// CHECK-NEXT: #define T8_71 int &&
// CHECK-NEXT:     T8_68 a1;
// CHECK-NEXT:     T8_69 a2;
// CHECK-NEXT:     T8_70 a3=a1;
// CHECK-NEXT:     T8_71 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_68 hiprandStatus_t
#define T8_69 hiprandStatus_t *
#define T8_70 hiprandStatus_t &
#define T8_71 hiprandStatus_t &&
    T8_68 a1;
    T8_69 a2;
    T8_70 a3=a1;
    T8_71 a4=std::move(a1);
}

{
// CHECK: #define T8_72 int
// CHECK-NEXT: #define T8_73 int *
// CHECK-NEXT: #define T8_74 int &
// CHECK-NEXT: #define T8_75 int &&
// CHECK-NEXT:     T8_72 a1;
// CHECK-NEXT:     T8_73 a2;
// CHECK-NEXT:     T8_74 a3=a1;
// CHECK-NEXT:     T8_75 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_72 hipfftResult_t
#define T8_73 hipfftResult_t *
#define T8_74 hipfftResult_t &
#define T8_75 hipfftResult_t &&
    T8_72 a1;
    T8_73 a2;
    T8_74 a3=a1;
    T8_75 a4=std::move(a1);
}

{
// CHECK: #define T8_76 dpct::queue_ptr
// CHECK-NEXT: #define T8_77 dpct::queue_ptr *
// CHECK-NEXT: #define T8_78 dpct::queue_ptr &
// CHECK-NEXT: #define T8_79 dpct::queue_ptr &&
// CHECK-NEXT:     T8_76 a1;
// CHECK-NEXT:     T8_77 a2;
// CHECK-NEXT:     T8_78 a3=a1;
// CHECK-NEXT:     T8_79 a4=std::move(a1);
// CHECK-NEXT: }
#define T8_76 hipStream_t
#define T8_77 hipStream_t *
#define T8_78 hipStream_t &
#define T8_79 hipStream_t &&
    T8_76 a1;
    T8_77 a2;
    T8_78 a3=a1;
    T8_79 a4=std::move(a1);
}

{
// CHECK: #define T8__2 sycl::queue *
// CHECK-NEXT: T8__2 a2;
#define T8__2 ihipStream_t *
  T8__2 a2;
}
}


// case 9
template <typename T> void template_foo(T var) {}
#define foo1(DataType) template_foo(DataType varname)
#define foo2(DataType) template_foo(DataType * varname)
#define foo3(DataType) template_foo(DataType & varname)
#define foo4(DataType) template_foo(DataType && varname)

// CHECK: template <> void foo1(sycl::range<3>){}
// CHECK-NEXT: template <> void foo2(sycl::range<3>){}
// CHECK-NEXT: template <> void foo3(sycl::range<3>){}
// CHECK-NEXT: template <> void foo4(sycl::range<3>){}
template <> void foo1(dim3){}
template <> void foo2(dim3){}
template <> void foo3(dim3){}
template <> void foo4(dim3){}

// CHECK: template <> void foo1(dpct::err0){}
// CHECK-NEXT: template <> void foo2(dpct::err0){}
// CHECK-NEXT: template <> void foo3(dpct::err0){}
// CHECK-NEXT: template <> void foo4(dpct::err0){}
template <> void foo1(hipError_t){}
template <> void foo2(hipError_t){}
template <> void foo3(hipError_t){}
template <> void foo4(hipError_t){}

// CHECK: template <> void foo1(int){}
// CHECK-NEXT: template <> void foo2(int){}
// CHECK-NEXT: template <> void foo3(int){}
// CHECK-NEXT: template <> void foo4(int){}
template <> void foo1(hipError_t){}
template <> void foo2(hipError_t){}
template <> void foo3(hipError_t){}
template <> void foo4(hipError_t){}

// CHECK: template <> void foo1(dpct::event_ptr){}
// CHECK-NEXT: template <> void foo2(dpct::event_ptr){}
// CHECK-NEXT: template <> void foo3(dpct::event_ptr){}
// CHECK-NEXT: template <> void foo4(dpct::event_ptr){}
template <> void foo1(hipEvent_t){}
template <> void foo2(hipEvent_t){}
template <> void foo3(hipEvent_t){}
template <> void foo4(hipEvent_t){}

// CHECK: template <> void foo1(dpct::blas::descriptor_ptr){}
// CHECK-NEXT: template <> void foo2(dpct::blas::descriptor_ptr){}
// CHECK-NEXT: template <> void foo3(dpct::blas::descriptor_ptr){}
// CHECK-NEXT: template <> void foo4(dpct::blas::descriptor_ptr){}
template <> void foo1(hipblasHandle_t){}
template <> void foo2(hipblasHandle_t){}
template <> void foo3(hipblasHandle_t){}
template <> void foo4(hipblasHandle_t){}

// CHECK: template <> void foo1(sycl::float2){}
// CHECK-NEXT: template <> void foo2(sycl::float2){}
// CHECK-NEXT: template <> void foo3(sycl::float2){}
// CHECK-NEXT: template <> void foo4(sycl::float2){}
template <> void foo1(hipComplex){}
template <> void foo2(hipComplex){}
template <> void foo3(hipComplex){}
template <> void foo4(hipComplex){}

// CHECK: template <> void foo1(sycl::double2){}
// CHECK-NEXT: template <> void foo2(sycl::double2){}
// CHECK-NEXT: template <> void foo3(sycl::double2){}
// CHECK-NEXT: template <> void foo4(sycl::double2){}
template <> void foo1(hipDoubleComplex){}
template <> void foo2(hipDoubleComplex){}
template <> void foo3(hipDoubleComplex){}
template <> void foo4(hipDoubleComplex){}

// CHECK: template <> void foo1(oneapi::mkl::uplo){}
// CHECK-NEXT: template <> void foo2(oneapi::mkl::uplo){}
// CHECK-NEXT: template <> void foo3(oneapi::mkl::uplo){}
// CHECK-NEXT: template <> void foo4(oneapi::mkl::uplo){}
template <> void foo1(hipblasFillMode_t){}
template <> void foo2(hipblasFillMode_t){}
template <> void foo3(hipblasFillMode_t){}
template <> void foo4(hipblasFillMode_t){}

// CHECK: template <> void foo1(oneapi::mkl::diag){}
// CHECK-NEXT: template <> void foo2(oneapi::mkl::diag){}
// CHECK-NEXT: template <> void foo3(oneapi::mkl::diag){}
// CHECK-NEXT: template <> void foo4(oneapi::mkl::diag){}
template <> void foo1(hipblasDiagType_t){}
template <> void foo2(hipblasDiagType_t){}
template <> void foo3(hipblasDiagType_t){}
template <> void foo4(hipblasDiagType_t){}

// CHECK: template <> void foo1(oneapi::mkl::side){}
// CHECK-NEXT: template <> void foo2(oneapi::mkl::side){}
// CHECK-NEXT: template <> void foo3(oneapi::mkl::side){}
// CHECK-NEXT: template <> void foo4(oneapi::mkl::side){}
template <> void foo1(hipblasSideMode_t){}
template <> void foo2(hipblasSideMode_t){}
template <> void foo3(hipblasSideMode_t){}
template <> void foo4(hipblasSideMode_t){}

// CHECK: template <> void foo1(oneapi::mkl::transpose){}
// CHECK-NEXT: template <> void foo2(oneapi::mkl::transpose){}
// CHECK-NEXT: template <> void foo3(oneapi::mkl::transpose){}
// CHECK-NEXT: template <> void foo4(oneapi::mkl::transpose){}
template <> void foo1(hipblasOperation_t){}
template <> void foo2(hipblasOperation_t){}
template <> void foo3(hipblasOperation_t){}
template <> void foo4(hipblasOperation_t){}

// CHECK: template <> void foo1(int){}
// CHECK-NEXT: template <> void foo2(int){}
// CHECK-NEXT: template <> void foo3(int){}
// CHECK-NEXT: template <> void foo4(int){}
template <> void foo1(hipblasStatus_t){}
template <> void foo2(hipblasStatus_t){}
template <> void foo3(hipblasStatus_t){}
template <> void foo4(hipblasStatus_t){}

// CHECK: template <> void foo1(int){}
// CHECK-NEXT: template <> void foo2(int){}
// CHECK-NEXT: template <> void foo3(int){}
// CHECK-NEXT: template <> void foo4(int){}
template <> void foo1(hipsolverStatus_t){}
template <> void foo2(hipsolverStatus_t){}
template <> void foo3(hipsolverStatus_t){}
template <> void foo4(hipsolverStatus_t){}

// CHECK: template <> void foo1(int64_t){}
// CHECK-NEXT: template <> void foo2(int64_t){}
// CHECK-NEXT: template <> void foo3(int64_t){}
// CHECK-NEXT: template <> void foo4(int64_t){}
template <> void foo1(hipsolverEigType_t){}
template <> void foo2(hipsolverEigType_t){}
template <> void foo3(hipsolverEigType_t){}
template <> void foo4(hipsolverEigType_t){}

// CHECK: template <> void foo1(oneapi::mkl::job){}
// CHECK-NEXT: template <> void foo2(oneapi::mkl::job){}
// CHECK-NEXT: template <> void foo3(oneapi::mkl::job){}
// CHECK-NEXT: template <> void foo4(oneapi::mkl::job){}
template <> void foo1(hipsolverEigMode_t){}
template <> void foo2(hipsolverEigMode_t){}
template <> void foo3(hipsolverEigMode_t){}
template <> void foo4(hipsolverEigMode_t){}

// CHECK: template <> void foo1(int){}
// CHECK-NEXT: template <> void foo2(int){}
// CHECK-NEXT: template <> void foo3(int){}
// CHECK-NEXT: template <> void foo4(int){}
template <> void foo1(hiprandStatus_t){}
template <> void foo2(hiprandStatus_t){}
template <> void foo3(hiprandStatus_t){}
template <> void foo4(hiprandStatus_t){}

// CHECK: template <> void foo1(int){}
// CHECK-NEXT: template <> void foo2(int){}
// CHECK-NEXT: template <> void foo3(int){}
// CHECK-NEXT: template <> void foo4(int){}
template <> void foo1(hipfftResult_t){}
template <> void foo2(hipfftResult_t){}
template <> void foo3(hipfftResult_t){}
template <> void foo4(hipfftResult_t){}

// CHECK: template <> void foo1(dpct::queue_ptr){}
// CHECK-NEXT: template <> void foo2(dpct::queue_ptr){}
// CHECK-NEXT: template <> void foo3(dpct::queue_ptr){}
// CHECK-NEXT: template <> void foo4(dpct::queue_ptr){}
template <> void foo1(hipStream_t){}
template <> void foo2(hipStream_t){}
template <> void foo3(hipStream_t){}
template <> void foo4(hipStream_t){}

void foo_struct(void) {
// CHECK: dpct::device_info d_t;
struct hipDeviceProp_t d_t;
}

// CHECK: void foo(dpct::queue_ptr& stream) {
// CHECK-NEXT:   dpct::queue_ptr s0;
// CHECK-NEXT:   dpct::queue_ptr &s1 = s0;
// CHECK-NEXT: }
void foo(hipStream_t& stream) {
  hipStream_t s0;
  hipStream_t &s1 = s0;
}
