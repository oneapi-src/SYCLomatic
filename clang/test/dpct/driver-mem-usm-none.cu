// RUN: dpct --usm-level=none --format-range=none -out-root %T/driver-mem-usm-none %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem-usm-none/driver-mem-usm-none.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)malloc(size);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);

    // CHECK: dpct::device_ptr f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: dpct::device_ptr f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = (dpct::device_ptr)dpct::dpct_malloc(size);
    hipMalloc(&f_D, size);

    // CHECK: dpct::queue_ptr stream;
    hipStream_t stream;
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, dpct::automatic, *stream);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, dpct::automatic);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(dpct::dpct_memcpy(f_D, f_A, size, dpct::automatic));
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, dpct::automatic, *stream);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, dpct::automatic);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: dpct::dpct_memcpy(f_A, f_D, size, dpct::automatic);
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    cuMemcpy(f_D, f_D2, size);
    // CHECK: CALL(dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    CALL(cuMemcpy(f_D, f_D2, size));
    // CHECK: r = DPCT_CHECK_ERROR(dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    r = cuMemcpy(f_D, f_D2, size);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream);
    cuMemcpyAsync(f_D, f_D2, size, stream);
    // CHECK: CALL(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream));
    CALL(cuMemcpyAsync(f_D, f_D2, size, stream));
    // CHECK: r = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream));
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    cuMemcpyAsync(f_D, f_D2, size, 0);
    // CHECK: CALL(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    CALL(cuMemcpyAsync(f_D, f_D2, size, 0));
    // CHECK: r = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    r = cuMemcpyAsync(f_D, f_D2, size, 0);

    // CHECK: dpct::memcpy_parameter cpy;
    // CHECK-NEXT: cpy.to.pitched.set_data_ptr(f_A);
    // CHECK-NEXT: cpy.to.pitched.set_pitch(20);
    // CHECK-NEXT: cpy.to.pos[1] = 10;
    // CHECK-NEXT: cpy.to.pos[0] = 15;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy.from.pitched.set_data_ptr(f_D);
    // CHECK-NEXT: cpy.from.pitched.set_pitch(20);
    // CHECK-NEXT: cpy.from.pos[1] = 10;
    // CHECK-NEXT: cpy.from.pos[0] = 15;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy.size[0] = 4;
    // CHECK-NEXT: cpy.size[1] = 7;
    hip_Memcpy2D cpy;
    cpy.dstMemoryType = hipMemoryTypeHost;
    cpy.dstHost = f_A;
    cpy.dstPitch = 20;
    cpy.dstY = 10;
    cpy.dstXInBytes = 15;

    cpy.srcMemoryType = hipMemoryTypeDevice;
    cpy.srcDevice = f_D;
    cpy.srcPitch = 20;
    cpy.srcY = 10;
    cpy.srcXInBytes = 15;

    cpy.WidthInBytes = 4;
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(cpy);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(cpy, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;
    size_t count = 32;
    // CHECK: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;
    hipDevice_t cudevice =0;
    hipError_t cu_err;

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, advise, cudevice);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, 0, cudevice);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, 0, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, (int)1, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, int(1), cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, static_cast<int>(1), cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = hipMemAdvise(devicePtr, count, 0, cudevice);
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, 0, cudevice);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    unsigned int v32 = 50000;
    unsigned short v16 = 20000;
    unsigned char v8 = (unsigned char) 200;
    //CHECK: dpct::dpct_memset_d32(f_D, v32, size);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, v16, size * 2);
    //CHECK-NEXT: dpct::dpct_memset(f_D, v8, size * 4);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, v32, size, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, v16, size * 2, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, v8, size * 4, *stream);
    //CHECK-NEXT: dpct::dpct_memset_d32(f_D, 1, v32, 4, 6);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, 1, v16, 4 * 2, 6);
    //CHECK-NEXT: dpct::dpct_memset(f_D, 1, v8, 4 * 4, 6);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, 1, v32, 4, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, 1, v16, 4 * 2, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, 1, v8, 4 * 4, 6, *stream);
    hipMemsetD32(f_D, v32, size);
    hipMemsetD16(f_D, v16, size * 2);
    hipMemsetD8(f_D, v8, size * 4);
    hipMemsetD32Async(f_D, v32, size, stream);
    hipMemsetD16Async(f_D, v16, size * 2, stream);
    hipMemsetD8Async(f_D, v8, size * 4, stream);
    hipMemsetD2D32(f_D, 1, v32, 4, 6);
    hipMemsetD2D16(f_D, 1, v16, 4 * 2, 6);
    hipMemsetD2D8(f_D, 1, v8, 4 * 4, 6);
    hipMemsetD2D32Async(f_D, 1, v32, 4, 6, stream);
    hipMemsetD2D16Async(f_D, 1, v16, 4 * 2, 6, stream);
    hipMemsetD2D8Async(f_D, 1, v8, 4 * 4, 6, stream);

    // CHECK: dpct::memcpy_parameter cpy2;
    // CHECK-EMPTY:
    // CHECK-NEXT: dpct::image_matrix_p ca;
    // CHECK-NEXT: cpy2.to.image = ca;
    // CHECK-NEXT: cpy2.to.pitched.set_pitch(5);
    // CHECK-NEXT: cpy2.to.pitched.set_y(4);
    // CHECK-NEXT: cpy2.to.pos[1] = 3;
    // CHECK-NEXT: cpy2.to.pos[2] = 2;
    // CHECK-NEXT: cpy2.to.pos[0] = 1;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy2.from.pitched.set_data_ptr(f_A);
    // CHECK-NEXT: cpy2.from.pitched.set_pitch(5);
    // CHECK-NEXT: cpy2.from.pitched.set_y(4);
    // CHECK-NEXT: cpy2.from.pos[1] = 3;
    // CHECK-NEXT: cpy2.from.pos[2] = 2;
    // CHECK-NEXT: cpy2.from.pos[0] = 1;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy2.size[0] = 3;
    // CHECK-NEXT: cpy2.size[1] = 2;
    // CHECK-NEXT: cpy2.size[2] = 1;
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    cpy2.dstMemoryType = hipMemoryTypeArray;
    cpy2.dstArray = ca;
    cpy2.dstPitch = 5;
    cpy2.dstHeight = 4;
    cpy2.dstY = 3;
    cpy2.dstZ = 2;
    cpy2.dstXInBytes = 1;
    cpy2.dstLOD = 0;

    cpy2.srcMemoryType = hipMemoryTypeHost;
    cpy2.srcHost = f_A;
    cpy2.srcPitch = 5;
    cpy2.srcHeight = 4;
    cpy2.srcY = 3;
    cpy2.srcZ = 2;
    cpy2.srcXInBytes = 1;
    cpy2.srcLOD = 0;

    cpy2.WidthInBytes = 3;
    cpy2.Height = 2;
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(cpy2);
    hipDrvMemcpy3D(&cpy2);

    hipStream_t cs;
    // CHECK: dpct::async_dpct_memcpy(cpy2, *cs);
    hipDrvMemcpy3DAsync(&cpy2, cs);

    // CHECK: dpct::dpct_free(f_D);
    hipFree(f_D);
    unsigned int flags;
    int host;
    float *h_A = (float *)malloc(100);

    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(flags = 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
