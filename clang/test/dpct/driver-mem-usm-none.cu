// RUN: dpct --usm-level=none --format-range=none -out-root %T/driver-mem-usm-none %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem-usm-none/driver-mem-usm-none.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)malloc(size);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);

    // CHECK: char * f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: char * f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = (char *)dpct::dpct_malloc(size);
    hipMalloc(&f_D, size);

    // CHECK: dpct::queue_ptr stream;
    hipStream_t stream;
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, dpct::automatic, *stream);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_D, f_A, size, dpct::automatic);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(dpct::dpct_memcpy(f_D, f_A, size, dpct::automatic));
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, dpct::automatic, *stream);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_A, f_D, size, dpct::automatic);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: dpct::dpct_memcpy(f_A, f_D, size, dpct::automatic);
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    cuMemcpy(f_D, f_D2, size);
    // CHECK: CALL(dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    CALL(cuMemcpy(f_D, f_D2, size));
    // CHECK: r = (dpct::dpct_memcpy(f_D, f_D2, size, dpct::automatic), 0);
    r = cuMemcpy(f_D, f_D2, size);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream);
    cuMemcpyAsync(f_D, f_D2, size, stream);
    // CHECK: CALL(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream));
    CALL(cuMemcpyAsync(f_D, f_D2, size, stream));
    // CHECK: r = (dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic, *stream), 0);
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    // CHECK: dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic);
    cuMemcpyAsync(f_D, f_D2, size, 0);
    // CHECK: CALL(dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic));
    CALL(cuMemcpyAsync(f_D, f_D2, size, 0));
    // CHECK: r = (dpct::async_dpct_memcpy(f_D, f_D2, size, dpct::automatic), 0);
    r = cuMemcpyAsync(f_D, f_D2, size, 0);

    // CHECK: dpct::pitched_data cpy_from_data_ct1, cpy_to_data_ct1;
    // CHECK: sycl::id<3> cpy_from_pos_ct1(0, 0, 0), cpy_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy_size_ct1(1, 1, 1);
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy_to_data_ct1.set_data_ptr(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy_to_data_ct1.set_pitch(20);
    cpy.dstPitch = 20;
    // CHECK: cpy_to_pos_ct1[1] = 10;
    cpy.dstY = 10;
    // CHECK: cpy_to_pos_ct1[0] = 15;
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy_from_data_ct1.set_data_ptr(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy_from_data_ct1.set_pitch(20);
    cpy.srcPitch = 20;
    // CHECK: cpy_from_pos_ct1[1] = 10;
    cpy.srcY = 10;
    // CHECK: cpy_from_pos_ct1[0] = 15;
    cpy.srcXInBytes = 15;

    // CHECK: cpy_size_ct1[0] = 4;
    cpy.WidthInBytes = 4;
    // CHECK: cpy_size_ct1[1] = 7;
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1, dpct::automatic, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;
    size_t count = 32;
    // CHECK: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;
    hipDevice_t cudevice =0;
    hipError_t cu_err;

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, advise, cudevice);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, 0, cudevice);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, 0, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, (int)1, cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, int(1), cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(hipMemAdvise(devicePtr, count, static_cast<int>(1), cudevice));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = hipMemAdvise(devicePtr, count, 0, cudevice);
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipMemAdvise(devicePtr, count, 0, cudevice);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: dpct::pitched_data cpy2_from_data_ct1, cpy2_to_data_ct1;
    // CHECK: sycl::id<3> cpy2_from_pos_ct1(0, 0, 0), cpy2_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy2_size_ct1(1, 1, 1);
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2_to_data_ct1 = ca->to_pitched_data();
    cpy2.dstArray = ca;
    // CHECK: cpy2_to_data_ct1.set_pitch(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2_to_data_ct1.set_y(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2_to_pos_ct1[1] = 3;
    cpy2.dstY = 3;
    // CHECK: cpy2_to_pos_ct1[2] = 2;
    cpy2.dstZ = 2;
    // CHECK: cpy2_to_pos_ct1[0] = 1;
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2_from_data_ct1.set_data_ptr(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2_from_data_ct1.set_pitch(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2_from_data_ct1.set_y(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2_from_pos_ct1[1] = 3;
    cpy2.srcY = 3;
    // CHECK: cpy2_from_pos_ct1[2] = 2;
    cpy2.srcZ = 2;
    // CHECK: cpy2_from_pos_ct1[0] = 1;
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2_size_ct1[0] = 3;
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2_size_ct1[1] = 2;
    cpy2.Height = 2;
    // CHECK: cpy2_size_ct1[2] = 1;
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1);
    hipDrvMemcpy3D(&cpy2);
    // CHECK: dpct::dpct_free(f_D);
    hipFree(f_D);
    unsigned int flags;
    int host;
    float *h_A = (float *)malloc(100);

    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError((flags = 0, 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
