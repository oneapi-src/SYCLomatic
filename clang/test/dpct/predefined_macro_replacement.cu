#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none --usm-level=none -out-root %T/predefined_macro_replacement %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -D__NVCC__ -D__CUDACC__
// RUN: FileCheck --input-file %T/predefined_macro_replacement/predefined_macro_replacement.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/predefined_macro_replacement/predefined_macro_replacement.dp.cpp -o %T/predefined_macro_replacement/predefined_macro_replacement.dp.o %}

//CHECK: #define DPCT_COMPAT_RT_VERSION {{[1-9][0-9]+}}

#include <stdio.h>
//CHECK: #ifdef DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT: void hello(const sycl::stream &[[STREAM:stream_ct1]]) { [[STREAM]] << "foo"; }
#ifdef __CUDA_ARCH__
__device__ void hello() { printf("foo"); }
#else
__device__ void hello() { printf("other"); }
#endif

//CHECK: #ifndef DPCT_COMPATIBILITY_TEMP
#ifndef __NVCC__
__device__ void hello2() { printf("hello2"); }
#endif
//CHECK: #if defined(SYCL_LANGUAGE_VERSION)
#if defined(__HIPCC__)
__device__ void hello3() { printf("hello2"); }
#endif

#if defined(xxx)
__device__ void hello4() { printf("hello2"); }
//CHECK: #elif defined(DPCT_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello4(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#elif defined(__CUDA_ARCH__)
__device__ void hello4() { printf("hello2"); }
#endif

#if defined(xxx)
__device__ void hello5() { printf("hello2"); }
//CHECK: #elif (DPCT_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT: void hello5(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#elif (__CUDA_ARCH__ >= 400)
__device__ void hello5() { printf("hello2"); }
#endif

//CHECK: #if defined(DPCT_COMPATIBILITY_TEMP)
//CHECK-NEXT: void hello6(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#if defined(__CUDA_ARCH__)
__device__ void hello6() { printf("hello2"); }
#endif

//CHECK: #ifndef DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT: __device__ void hello7() { printf("hello2"); }
//CHECK-NEXT: #else
//CHECK-NEXT: void hello7(const sycl::stream &[[STREAM]]) { [[STREAM]] << "hello2"; }
#ifndef __CUDA_ARCH__
__device__ void hello7() { printf("hello2"); }
#else
__device__ void hello7() { printf("hello2"); }
#endif

__global__ void hello8() {}

__device__ void test(){
//CHECK:#if (DPCT_COMPATIBILITY_TEMP >= 400) &&  (DPCT_COMPATIBILITY_TEMP >= 400)
//CHECK-NEXT:[[STREAM]] << ">400, \n";
//CHECK-NEXT:#elif (DPCT_COMPATIBILITY_TEMP >200)
//CHECK-NEXT:printf(">200, \n");
//CHECK-NEXT:#else
//CHECK-NEXT:printf("<200, \n");
//CHECK-NEXT:#endif
#if (__CUDA_ARCH__ >= 400) &&  (__CUDA_ARCH__ >= 400)
printf(">400, \n");
#elif (__CUDA_ARCH__ >200)
printf(">200, \n");
#else
printf("<200, \n");
#endif
}


int main() {
//CHECK: #if defined(DPCT_COMPATIBILITY_TEMP)
//CHECK-NEXT:     q_ct1.parallel_for(
#if defined(__NVCC__)
  hello8<<<1,1>>>();
#else
  hello();
#endif

//CHECK: #ifdef DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT:     q_ct1.parallel_for(
  #ifdef __NVCC__
  hello8<<<1,1>>>();
#else
  hello();
#endif

//CHECK: #if DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT:     q_ct1.parallel_for(
  #if __NVCC__
  hello8<<<1,1>>>();
#else
  hello();
#endif
  return 0;
}

//CHECK: #define AAA DPCT_COMPATIBILITY_TEMP
//CHECK-NEXT: #define BBB SYCL_LANGUAGE_VERSION
//CHECK-NEXT: #define CCC DPCT_COMPATIBILITY_TEMP
#define AAA __CUDA_ARCH__
#define BBB __HIPCC__
#define CCC __NVCC__

//CHECK: #ifdef __DPCT_HPP__
//CHECK-NEXT:#endif
//CHECK-NEXT:#ifdef __DPCT_HPP__
//CHECK-NEXT:#endif
#ifdef __DRIVER_TYPES_H__
#endif
#ifdef __CUDA_RUNTIME_H__
#endif

//CHECK: #if defined(__DPCT_HPP__)
//CHECK-NEXT:#endif
//CHECK-NEXT:#if defined(__DPCT_HPP__)
//CHECK-NEXT:#endif
#if defined(__DRIVER_TYPES_H__)
#endif
#if defined(__CUDA_RUNTIME_H__)
#endif

int foo(int num) {
//CHECK: #if DPCT_COMPAT_RT_VERSION >= 4000
//CHECK-NEXT: dpct::get_current_device().reset();
//CHECK-NEXT: #else
//CHECK-NEXT: hipDeviceReset();
//CHECK-NEXT: #endif
#if CUDART_VERSION >= 4000
  hipDeviceReset();
#else
  hipDeviceReset();
#endif

  return 0;

}

int foo1() {
//CHECK: #ifdef DPCT_COMPAT_RT_VERSION
//CHECK-NEXT: sycl::int2 a;
//CHECK-NEXT: #endif
//CHECK-NEXT: #ifndef DPCT_COMPAT_RT_VERSION
//CHECK-NEXT: int2 b;
//CHECK-NEXT: #endif
#ifdef CUDART_VERSION
int2 a;
#endif
#ifndef CUDART_VERSION
int2 b;
#endif
  a.x = 1;
  return a.x;
}

int foo2(){
  int version;
  //CHECK: int ret = DPCT_CHECK_ERROR(version =  dpct::get_major_version(dpct::get_current_device()));
  int ret = hipRuntimeGetVersion(&version);
  int major = version / 1000;
  int minor = (version - major * 1000) / 10;
  int pl = version - major * 1000 - minor * 10;
  //CHECK: if (version != DPCT_COMPAT_RT_VERSION) {
  //CHECK-NEXT:   major = DPCT_COMPAT_RT_VERSION / 1000;
  //CHECK-NEXT:   minor = (DPCT_COMPAT_RT_VERSION - major * 1000) / 10;
  //CHECK-NEXT:   pl = DPCT_COMPAT_RT_VERSION - major * 1000 - minor * 10;
  //CHECK-NEXT: }
  if (version != CUDART_VERSION) {
    major = CUDART_VERSION / 1000;
    minor = (CUDART_VERSION - major * 1000) / 10;
    pl = CUDART_VERSION - major * 1000 - minor * 10;
  }
}

#define AAAA 1
//CHECK: void foo3() {
//CHECK-NEXT: #if defined DPCT_COMPAT_RT_VERSION && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   sycl::int2 a1;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if defined(DPCT_COMPAT_RT_VERSION) && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   sycl::int2 a2;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if (DPCT_COMPAT_RT_VERSION >= 4000) && AAAA
//CHECK-NEXT:   sycl::int2 a3;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if !(DPCT_COMPAT_RT_VERSION > 4000) && AAAA
//CHECK-NEXT:   int2 a4;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if (DPCT_COMPAT_RT_VERSION > 4000 ? 1 : 0) && AAAA
//CHECK-NEXT:   sycl::int2 a5;
//CHECK-NEXT: #endif
//CHECK-NEXT: }
void foo3() {
#if defined CUDART_VERSION && (CUDART_VERSION >= 4000)
  int2 a1;
#endif
#if defined(CUDART_VERSION) && (CUDART_VERSION >= 4000)
  int2 a2;
#endif
#if (CUDART_VERSION >= 4000) && AAAA
  int2 a3;
#endif
#if !(CUDART_VERSION > 4000) && AAAA
  int2 a4;
#endif
#if (CUDART_VERSION > 4000 ? 1 : 0) && AAAA
  int2 a5;
#endif
}

//CHECK: void foo4() {
//CHECK-NEXT: #define BBBB 0
//CHECK-NEXT: #if BBBB
//CHECK-NEXT: #elif defined DPCT_COMPAT_RT_VERSION && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   sycl::int2 a1;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if BBBB
//CHECK-NEXT: #elif defined(DPCT_COMPAT_RT_VERSION) && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   sycl::int2 a2;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if BBBB
//CHECK-NEXT: #elif (DPCT_COMPAT_RT_VERSION >= 4000) && AAAA
//CHECK-NEXT:   sycl::int2 a3;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if BBBB
//CHECK-NEXT: #elif !(DPCT_COMPAT_RT_VERSION > 4000) && AAAA
//CHECK-NEXT:   int2 a4;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if BBBB
//CHECK-NEXT: #elif (DPCT_COMPAT_RT_VERSION > 4000 ? 1 : 0) && AAAA
//CHECK-NEXT:   sycl::int2 a5;
//CHECK-NEXT: #endif
//CHECK-NEXT: }
void foo4() {
#define BBBB 0
#if BBBB
#elif defined CUDART_VERSION && (CUDART_VERSION >= 4000)
  int2 a1;
#endif
#if BBBB
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 4000)
  int2 a2;
#endif
#if BBBB
#elif (CUDART_VERSION >= 4000) && AAAA
  int2 a3;
#endif
#if BBBB
#elif !(CUDART_VERSION > 4000) && AAAA
  int2 a4;
#endif
#if BBBB
#elif (CUDART_VERSION > 4000 ? 1 : 0) && AAAA
  int2 a5;
#endif
}
#undef BBBB

//CHECK: void foo5() {
//CHECK-NEXT: #define CCCC 1
//CHECK-NEXT: #if CCCC
//CHECK-NEXT: #elif defined DPCT_COMPAT_RT_VERSION && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   int2 a1;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if CCCC
//CHECK-NEXT: #elif defined(DPCT_COMPAT_RT_VERSION) && (DPCT_COMPAT_RT_VERSION >= 4000)
//CHECK-NEXT:   int2 a2;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if CCCC
//CHECK-NEXT: #elif (DPCT_COMPAT_RT_VERSION >= 4000) && AAAA
//CHECK-NEXT:   int2 a3;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if CCCC
//CHECK-NEXT: #elif !(DPCT_COMPAT_RT_VERSION > 4000) && AAAA
//CHECK-NEXT:   int2 a4;
//CHECK-NEXT: #endif
//CHECK-NEXT: #if CCCC
//CHECK-NEXT: #elif (DPCT_COMPAT_RT_VERSION > 4000 ? 1 : 0) && AAAA
//CHECK-NEXT:   int2 a5;
//CHECK-NEXT: #endif
//CHECK-NEXT: }
void foo5() {
#define CCCC 1
#if CCCC
#elif defined CUDART_VERSION && (CUDART_VERSION >= 4000)
  int2 a1;
#endif
#if CCCC
#elif defined(CUDART_VERSION) && (CUDART_VERSION >= 4000)
  int2 a2;
#endif
#if CCCC
#elif (CUDART_VERSION >= 4000) && AAAA
  int2 a3;
#endif
#if CCCC
#elif !(CUDART_VERSION > 4000) && AAAA
  int2 a4;
#endif
#if CCCC
#elif (CUDART_VERSION > 4000 ? 1 : 0) && AAAA
  int2 a5;
#endif
}
#undef CCCC
#undef AAAA

//CHECK: void foo6() {
//CHECK-NEXT:   float *f;
//CHECK-NEXT: #if (DPCT_COMPAT_RT_VERSION >= 12000)
void foo6() {
  float *f;
#if (CUDART_VERSION >= 12000)
  hipMalloc(&f, 4);
#else
  hipHostMalloc(&f, 4);
#endif
}

//CHECK: void foo7() {
//CHECK-NEXT:   float *f;
//CHECK-NEXT: #if (DPCT_COMPAT_RT_VERSION >= 12000)
void foo7() {
  float *f;
#if (__CUDART_API_VERSION >= 12000)
  hipMalloc(&f, 4);
#else
  hipHostMalloc(&f, 4);
#endif
}

//CHECK: void foo8() {
//CHECK-NEXT:   float *f;
//CHECK-NEXT: #if (DPCT_COMPAT_RT_VERSION >= 12000)
void foo8() {
  float *f;
#if (CUDA_VERSION >= 12000)
  hipMalloc(&f, 4);
#else
  hipHostMalloc(&f, 4);
#endif
}
