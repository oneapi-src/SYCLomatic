
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/cudaStream_init_test %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cudaStream_init_test/cudaStream_init_test.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>

int main(){
  // CHECK: dpct::queue_ptr s0, s1{&dpct::get_default_queue()};
  hipStream_t  s0, s1{0};

  // CHECK: dpct::queue_ptr s2{&dpct::get_default_queue()};
  hipStream_t s2{0};

  // CHECK: s0 = dpct::get_current_device().create_queue();
  hipStreamCreate(&s0);

  // CHECK: dpct::queue_ptr s3(&dpct::get_default_queue());
  hipStream_t s3(0);

  // CHECK: dpct::queue_ptr s4 = &dpct::get_default_queue();
  hipStream_t s4 = 0;

  // CHECK: dpct::queue_ptr s5, s6(&dpct::get_default_queue()), s7 = &dpct::get_default_queue();
  hipStream_t s5, s6(0), s7 = 0;

  // CHECK: dpct::queue_ptr s8 = &dpct::get_default_queue();
  hipStream_t s8 = NULL;

  struct a{
    // CHECK: dpct::queue_ptr s9;
    hipStream_t s9;
    // CHECK: dpct::queue_ptr s10 = &dpct::get_default_queue(), s11, s12{&dpct::get_default_queue()};
    hipStream_t s10 = 0, s11, s12{0};
  };

}


