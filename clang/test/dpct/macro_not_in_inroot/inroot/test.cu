// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" --in-root %S --extra-arg="-I  %S/.."
// RUN: FileCheck --input-file %T/test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/test.dp.cpp -o %T/test.dp.o %}
#ifndef  NO_BUILD_TEST
#include "outer/macro_def.h"
#include "hip/hip_runtime.h"

void foo() {}

// CHECK: #define MACRO_B \
// CHECK-NEXT: foo();\
// CHECK-NEXT: MACRO(dpct::get_error_string_dummy({{[0-9]+}}));
#define MACRO_B \
foo();\
MACRO(hipGetErrorString(hipErrorInvalidValue));

int main() {
  // CHECK: MACRO_B
  MACRO_B
  return 0;
}
#endif
