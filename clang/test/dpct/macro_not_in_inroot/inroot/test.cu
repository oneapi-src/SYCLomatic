// RUN: dpct --format-range=none --out-root %T %s --cuda-include-path="%cuda-path/include" --in-root %S --extra-arg="-I  %S/.."
// RUN: FileCheck --input-file %T/test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/test.dp.cpp -o %T/test.dp.o %}
#ifndef  BUILD_TEST
#include "outer/macro_def.h"
#include "hip/hip_runtime.h"

void foo() {}

// CHECK: #define MACRO_B \
// CHECK-NEXT: foo();\
// CHECK-NEXT: MACRO(dpct::get_error_string_dummy(1));
#define MACRO_B \
foo();\
MACRO(hipGetErrorString(hipErrorInvalidValue));

int main() {
  // CHECK: MACRO_B
  MACRO_B
  return 0;
}
#endif
