// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasTtrmm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasTtrmm/cublasTtrmm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int foo1();
hipDoubleComplex * foo2();


int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;

  int side0 = 0; int side1 = 1; int fill0 = 0; int fill1 = 1;
  int trans0 = 0; int trans1 = 1; int trans2 = 2; int diag0 = 0; int diag1 = 1;
  // CHECK: status = DPCT_CHECK_ERROR(dpct::trmm(*handle, (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc));
  // CHECK-NEXT: dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);
  status = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);
  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, B_S, ldb, C_S, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::trmm(*handle, (oneapi::mkl::side)side1, fill1 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), (oneapi::mkl::diag)diag1, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc));
  // CHECK-NEXT: dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);
  status = hipblasDtrmm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, (hipblasDiagType_t)diag1, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);
  hipblasDtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, B_D, ldb, C_D, ldc);


  const hipComplex *A_C = 0;
  const hipComplex *B_C = 0;
  hipComplex *C_C = 0;
  hipComplex alpha_C = make_hipComplex(1.0f,0.0f);
  const hipDoubleComplex *A_Z = 0;
  const hipDoubleComplex *B_Z = 0;
  hipDoubleComplex *C_Z = 0;
  hipDoubleComplex alpha_Z = make_hipDoubleComplex(1.0,0.0);


  // CHECK: status = DPCT_CHECK_ERROR(dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, dpct::get_transpose(trans2), oneapi::mkl::diag::nonunit, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc));
  // CHECK-NEXT: dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);
  status = hipblasCtrmm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)0, (hipblasOperation_t)trans2, (hipblasDiagType_t)0, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);
  hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_C, A_C, lda, B_C, ldb, C_C, ldc);


  // CHECK: status = DPCT_CHECK_ERROR(dpct::trmm(*handle, oneapi::mkl::side::right, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, oneapi::mkl::diag::unit, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc));
  // CHECK-NEXT: dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);
  status = hipblasZtrmm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)1, (hipblasOperation_t)2, (hipblasDiagType_t)1, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, C_Z, ldc);


  // CHECK: dpct::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, dpct::get_transpose(foo1()), oneapi::mkl::diag::nonunit, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, foo2(), ldc);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, (hipblasOperation_t)foo1(), HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_Z, A_Z, lda, B_Z, ldb, foo2(), ldc);
}

