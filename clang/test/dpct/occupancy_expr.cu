
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/occupancy_expr %s --use-experimental-features=occupancy-calculation --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/occupancy_expr/occupancy_expr.dp.cpp

__global__ void k() {}

int main() {
  int num_blocks;
  int block_size = 128;
  size_t dynamic_shared_memory_size = 0 ;
  // CHECK: dpct::experimental::sycl_occupancy_max_active_wg_per_xecore(&num_blocks, block_size, dynamic_shared_memory_size + dpct_placeholder /* static shared local memory size */);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, k, block_size, dynamic_shared_memory_size);
  return 0;
}
