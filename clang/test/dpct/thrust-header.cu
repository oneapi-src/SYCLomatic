// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/thrust-header %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: FileCheck --input-file %T/thrust-header/thrust-header.dp.cpp --match-full-lines %s
// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT: #include <oneapi/dpl/algorithm>
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK-NEXT: #include <algorithm>
#include <cstdio>
#include <algorithm>
// CHECK: #include <dpct/dpl_utils.hpp>
#include <hip/hip_runtime.h>
#include <thrust/copy.h>
#include <thrust/sequence.h>
#include <thrust/sort.h>
#include <thrust/device_ptr.h>
int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  int *mapsp1D, *mapspkeyD,*mapspvalD;
  int numsH=10;

  hipMalloc(&mapsp1D, numsH*sizeof(int));
  hipMalloc(&mapspkeyD, numsH*sizeof(int));
  hipMalloc(&mapspvalD, numsH*sizeof(int));

// CHECK:  dpct::device_pointer<int> mapsp1T(mapsp1D);
  thrust::device_ptr<int> mapsp1T(mapsp1D);
// CHECK:  dpct::device_pointer<int> mapspkeyT(mapspkeyD);
  thrust::device_ptr<int> mapspkeyT(mapspkeyD);
// CHECK:  dpct::device_pointer<int> mapspvalT(mapspvalD);
  thrust::device_ptr<int> mapspvalT(mapspvalD);

// CHECK:  std::copy(oneapi::dpl::execution::make_device_policy(q_ct1), mapsp1T, mapsp1T + numsH, mapspkeyT);
  thrust::copy(mapsp1T, mapsp1T + numsH, mapspkeyT);
// CHECK:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), mapspkeyT, mapspkeyT + numsH, mapspvalT);
  thrust::stable_sort_by_key(mapspkeyT, mapspkeyT + numsH, mapspvalT);
}

