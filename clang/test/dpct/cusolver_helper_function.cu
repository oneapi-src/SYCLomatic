// RUN: dpct --format-range=none -out-root %T/cusolver_helper_function %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolver_helper_function/cusolver_helper_function.dp.cpp --match-full-lines %s

//CHECK:#include <sycl/sycl.hpp>
//CHECK-NEXT:#include <dpct/dpct.hpp>
//CHECK-NEXT:#include <oneapi/mkl.hpp>
//CHECK-NEXT:#include <dpct/lapack_utils.hpp>

#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  float *a_s, *b_s, *w_s, *work_s;
  int lwork_s;
  int *devInfo;
  //CHECK:status = (lwork_s = oneapi::mkl::lapack::sygvd_scratchpad_size(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 3, 3, 3), 0);
  hipsolverStatus_t status;
  status = hipsolverDnSsygvd_bufferSize(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s, 3, b_s, 3, w_s, &lwork_s);
  //CHECK:status = (dpct::lapack::sygvd(*handle, 1, oneapi::mkl::job::vec, oneapi::mkl::uplo::upper, 3, a_s, 3, b_s, 3, w_s, work_s, lwork_s, devInfo), 0);
  status = hipsolverDnSsygvd(handle, HIPSOLVER_EIG_TYPE_1, HIPSOLVER_EIG_MODE_VECTOR, HIPBLAS_FILL_MODE_UPPER, 3, a_s, 3, b_s, 3, w_s, work_s, lwork_s, devInfo);
  return 0;
}
