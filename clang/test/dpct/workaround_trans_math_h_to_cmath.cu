// RUN: dpct --format-range=none -out-root %T/workaround_trans_math_h_to_cmath %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/workaround_trans_math_h_to_cmath/workaround_trans_math_h_to_cmath.dp.cpp --match-full-lines %s
//CHECK:#include <sycl/sycl.hpp>
//CHECK-NEXT:#include <dpct/dpct.hpp>
//CHECK://math header
//CHECK: #include <math.h>
//CHECK-NEXT: #include <cmath>
//CHECK-NEXT: #include "math.h"
#include <hip/hip_runtime.h>
//math header
#include <math.h>
#include <cmath>
#include "math.h"

