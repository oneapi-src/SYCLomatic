#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none --usm-level=none -out-root %T/vector_type_cub %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/vector_type_cub/vector_type_cub.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/vector_type_cub/vector_type_cub.dp.cpp -o %T/vector_type_cub/vector_type_cub.dp.o %}

#ifndef BUILD_TEST
#include <hipcub/hipcub.hpp>

__device__ char1 operator+(char1 a, char1 b) {
  // CHECK: return char(a + b);
  return make_char1(a.x + b.x);
}

__global__ void test_make_char1() {
  typedef hipcub::BlockReduce<char1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: char res = sycl::reduce_over_group({{.+}}, char(1.), {{.+}});
  char1 res = BlockReduce(smem_storage).Sum(make_char1(1.));
}

__device__ char2 operator+(char2 a, char2 b) {
  // CHECK: return sycl::char2(a.x() + b.x(), a.y() + b.y());
  return make_char2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_char2() {
  typedef hipcub::BlockReduce<char2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::char2 res = sycl::reduce_over_group({{.+}}, sycl::char2(1., 2.), {{.+}});
  char2 res = BlockReduce(smem_storage).Sum(make_char2(1., 2.));
}

__device__ char3 operator+(char3 a, char3 b) {
  // CHECK: return sycl::char3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_char3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_char3() {
  typedef hipcub::BlockReduce<char3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::char3 res = sycl::reduce_over_group({{.+}}, sycl::char3(1., 2., 3.), {{.+}});
  char3 res = BlockReduce(smem_storage).Sum(make_char3(1., 2., 3.));
}

__device__ char4 operator+(char4 a, char4 b) {
  // CHECK: return sycl::char4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_char4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_char4() {
  typedef hipcub::BlockReduce<char4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::char4 res = sycl::reduce_over_group({{.+}}, sycl::char4(1., 2., 3., 4.), {{.+}});
  char4 res = BlockReduce(smem_storage).Sum(make_char4(1., 2., 3., 4.));
}

__device__ uchar1 operator+(uchar1 a, uchar1 b) {
  // CHECK: return uint8_t(a + b);
  return make_uchar1(a.x + b.x);
}

__global__ void test_make_uchar1() {
  typedef hipcub::BlockReduce<uchar1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: uint8_t res = sycl::reduce_over_group({{.+}}, uint8_t(1.), {{.+}});
  uchar1 res = BlockReduce(smem_storage).Sum(make_uchar1(1.));
}

__device__ uchar2 operator+(uchar2 a, uchar2 b) {
  // CHECK: return sycl::uchar2(a.x() + b.x(), a.y() + b.y());
  return make_uchar2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_uchar2() {
  typedef hipcub::BlockReduce<uchar2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uchar2 res = sycl::reduce_over_group({{.+}}, sycl::uchar2(1., 2.), {{.+}});
  uchar2 res = BlockReduce(smem_storage).Sum(make_uchar2(1., 2.));
}

__device__ uchar3 operator+(uchar3 a, uchar3 b) {
  // CHECK: return sycl::uchar3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_uchar3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_uchar3() {
  typedef hipcub::BlockReduce<uchar3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uchar3 res = sycl::reduce_over_group({{.+}}, sycl::uchar3(1., 2., 3.), {{.+}});
  uchar3 res = BlockReduce(smem_storage).Sum(make_uchar3(1., 2., 3.));
}

__device__ uchar4 operator+(uchar4 a, uchar4 b) {
  // CHECK: return sycl::uchar4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_uchar4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_uchar4() {
  typedef hipcub::BlockReduce<uchar4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uchar4 res = sycl::reduce_over_group({{.+}}, sycl::uchar4(1., 2., 3., 4.), {{.+}});
  uchar4 res = BlockReduce(smem_storage).Sum(make_uchar4(1., 2., 3., 4.));
}

__device__ short1 operator+(short1 a, short1 b) {
  // CHECK: return short(a + b);
  return make_short1(a.x + b.x);
}

__global__ void test_make_short1() {
  typedef hipcub::BlockReduce<short1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: short res = sycl::reduce_over_group({{.+}}, short(1.), {{.+}});
  short1 res = BlockReduce(smem_storage).Sum(make_short1(1.));
}

__device__ short2 operator+(short2 a, short2 b) {
  // CHECK: return sycl::short2(a.x() + b.x(), a.y() + b.y());
  return make_short2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_short2() {
  typedef hipcub::BlockReduce<short2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::short2 res = sycl::reduce_over_group({{.+}}, sycl::short2(1., 2.), {{.+}});
  short2 res = BlockReduce(smem_storage).Sum(make_short2(1., 2.));
}

__device__ short3 operator+(short3 a, short3 b) {
  // CHECK: return sycl::short3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_short3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_short3() {
  typedef hipcub::BlockReduce<short3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::short3 res = sycl::reduce_over_group({{.+}}, sycl::short3(1., 2., 3.), {{.+}});
  short3 res = BlockReduce(smem_storage).Sum(make_short3(1., 2., 3.));
}

__device__ short4 operator+(short4 a, short4 b) {
  // CHECK: return sycl::short4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_short4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_short4() {
  typedef hipcub::BlockReduce<short4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::short4 res = sycl::reduce_over_group({{.+}}, sycl::short4(1., 2., 3., 4.), {{.+}});
  short4 res = BlockReduce(smem_storage).Sum(make_short4(1., 2., 3., 4.));
}

__device__ ushort1 operator+(ushort1 a, ushort1 b) {
  // CHECK: return uint16_t(a + b);
  return make_ushort1(a.x + b.x);
}

__global__ void test_make_ushort1() {
  typedef hipcub::BlockReduce<ushort1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: uint16_t res = sycl::reduce_over_group({{.+}}, uint16_t(1.), {{.+}});
  ushort1 res = BlockReduce(smem_storage).Sum(make_ushort1(1.));
}

__device__ ushort2 operator+(ushort2 a, ushort2 b) {
  // CHECK: return sycl::ushort2(a.x() + b.x(), a.y() + b.y());
  return make_ushort2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_ushort2() {
  typedef hipcub::BlockReduce<ushort2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ushort2 res = sycl::reduce_over_group({{.+}}, sycl::ushort2(1., 2.), {{.+}});
  ushort2 res = BlockReduce(smem_storage).Sum(make_ushort2(1., 2.));
}

__device__ ushort3 operator+(ushort3 a, ushort3 b) {
  // CHECK: return sycl::ushort3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_ushort3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_ushort3() {
  typedef hipcub::BlockReduce<ushort3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ushort3 res = sycl::reduce_over_group({{.+}}, sycl::ushort3(1., 2., 3.), {{.+}});
  ushort3 res = BlockReduce(smem_storage).Sum(make_ushort3(1., 2., 3.));
}

__device__ ushort4 operator+(ushort4 a, ushort4 b) {
  // CHECK: return sycl::ushort4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_ushort4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_ushort4() {
  typedef hipcub::BlockReduce<ushort4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ushort4 res = sycl::reduce_over_group({{.+}}, sycl::ushort4(1., 2., 3., 4.), {{.+}});
  ushort4 res = BlockReduce(smem_storage).Sum(make_ushort4(1., 2., 3., 4.));
}

__device__ int1 operator+(int1 a, int1 b) {
  // CHECK: return int(a + b);
  return make_int1(a.x + b.x);
}

__global__ void test_make_int1() {
  typedef hipcub::BlockReduce<int1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: int res = sycl::reduce_over_group({{.+}}, int(1.), {{.+}});
  int1 res = BlockReduce(smem_storage).Sum(make_int1(1.));
}

__device__ int2 operator+(int2 a, int2 b) {
  // CHECK: return sycl::int2(a.x() + b.x(), a.y() + b.y());
  return make_int2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_int2() {
  typedef hipcub::BlockReduce<int2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::int2 res = sycl::reduce_over_group({{.+}}, sycl::int2(1., 2.), {{.+}});
  int2 res = BlockReduce(smem_storage).Sum(make_int2(1., 2.));
}

__device__ int3 operator+(int3 a, int3 b) {
  // CHECK: return sycl::int3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_int3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_int3() {
  typedef hipcub::BlockReduce<int3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::int3 res = sycl::reduce_over_group({{.+}}, sycl::int3(1., 2., 3.), {{.+}});
  int3 res = BlockReduce(smem_storage).Sum(make_int3(1., 2., 3.));
}

__device__ int4 operator+(int4 a, int4 b) {
  // CHECK: return sycl::int4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_int4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_int4() {
  typedef hipcub::BlockReduce<int4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::int4 res = sycl::reduce_over_group({{.+}}, sycl::int4(1., 2., 3., 4.), {{.+}});
  int4 res = BlockReduce(smem_storage).Sum(make_int4(1., 2., 3., 4.));
}

__device__ uint1 operator+(uint1 a, uint1 b) {
  // CHECK: return uint32_t(a + b);
  return make_uint1(a.x + b.x);
}

__global__ void test_make_uint1() {
  typedef hipcub::BlockReduce<uint1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: uint32_t res = sycl::reduce_over_group({{.+}}, uint32_t(1.), {{.+}});
  uint1 res = BlockReduce(smem_storage).Sum(make_uint1(1.));
}

__device__ uint2 operator+(uint2 a, uint2 b) {
  // CHECK: return sycl::uint2(a.x() + b.x(), a.y() + b.y());
  return make_uint2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_uint2() {
  typedef hipcub::BlockReduce<uint2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uint2 res = sycl::reduce_over_group({{.+}}, sycl::uint2(1., 2.), {{.+}});
  uint2 res = BlockReduce(smem_storage).Sum(make_uint2(1., 2.));
}

__device__ uint3 operator+(uint3 a, uint3 b) {
  // CHECK: return sycl::uint3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_uint3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_uint3() {
  typedef hipcub::BlockReduce<uint3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uint3 res = sycl::reduce_over_group({{.+}}, sycl::uint3(1., 2., 3.), {{.+}});
  uint3 res = BlockReduce(smem_storage).Sum(make_uint3(1., 2., 3.));
}

__device__ uint4 operator+(uint4 a, uint4 b) {
  // CHECK: return sycl::uint4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_uint4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_uint4() {
  typedef hipcub::BlockReduce<uint4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::uint4 res = sycl::reduce_over_group({{.+}}, sycl::uint4(1., 2., 3., 4.), {{.+}});
  uint4 res = BlockReduce(smem_storage).Sum(make_uint4(1., 2., 3., 4.));
}

__device__ long1 operator+(long1 a, long1 b) {
  // CHECK: return long(a + b);
  return make_long1(a.x + b.x);
}

__global__ void test_make_long1() {
  typedef hipcub::BlockReduce<long1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: long res = sycl::reduce_over_group({{.+}}, long(1.), {{.+}});
  long1 res = BlockReduce(smem_storage).Sum(make_long1(1.));
}

__device__ long2 operator+(long2 a, long2 b) {
  // CHECK: return sycl::long2(a.x() + b.x(), a.y() + b.y());
  return make_long2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_long2() {
  typedef hipcub::BlockReduce<long2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::long2 res = sycl::reduce_over_group({{.+}}, sycl::long2(1., 2.), {{.+}});
  long2 res = BlockReduce(smem_storage).Sum(make_long2(1., 2.));
}

__device__ long3 operator+(long3 a, long3 b) {
  // CHECK: return sycl::long3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_long3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_long3() {
  typedef hipcub::BlockReduce<long3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::long3 res = sycl::reduce_over_group({{.+}}, sycl::long3(1., 2., 3.), {{.+}});
  long3 res = BlockReduce(smem_storage).Sum(make_long3(1., 2., 3.));
}

__device__ long4 operator+(long4 a, long4 b) {
  // CHECK: return sycl::long4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_long4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_long4() {
  typedef hipcub::BlockReduce<long4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::long4 res = sycl::reduce_over_group({{.+}}, sycl::long4(1., 2., 3., 4.), {{.+}});
  long4 res = BlockReduce(smem_storage).Sum(make_long4(1., 2., 3., 4.));
}

__device__ ulong1 operator+(ulong1 a, ulong1 b) {
  // CHECK: return uint64_t(a + b);
  return make_ulong1(a.x + b.x);
}

__global__ void test_make_ulong1() {
  typedef hipcub::BlockReduce<ulong1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: uint64_t res = sycl::reduce_over_group({{.+}}, uint64_t(1.), {{.+}});
  ulong1 res = BlockReduce(smem_storage).Sum(make_ulong1(1.));
}

__device__ ulong2 operator+(ulong2 a, ulong2 b) {
  // CHECK: return sycl::ulong2(a.x() + b.x(), a.y() + b.y());
  return make_ulong2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_ulong2() {
  typedef hipcub::BlockReduce<ulong2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ulong2 res = sycl::reduce_over_group({{.+}}, sycl::ulong2(1., 2.), {{.+}});
  ulong2 res = BlockReduce(smem_storage).Sum(make_ulong2(1., 2.));
}

__device__ ulong3 operator+(ulong3 a, ulong3 b) {
  // CHECK: return sycl::ulong3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_ulong3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_ulong3() {
  typedef hipcub::BlockReduce<ulong3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ulong3 res = sycl::reduce_over_group({{.+}}, sycl::ulong3(1., 2., 3.), {{.+}});
  ulong3 res = BlockReduce(smem_storage).Sum(make_ulong3(1., 2., 3.));
}

__device__ ulong4 operator+(ulong4 a, ulong4 b) {
  // CHECK: return sycl::ulong4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_ulong4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_ulong4() {
  typedef hipcub::BlockReduce<ulong4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::ulong4 res = sycl::reduce_over_group({{.+}}, sycl::ulong4(1., 2., 3., 4.), {{.+}});
  ulong4 res = BlockReduce(smem_storage).Sum(make_ulong4(1., 2., 3., 4.));
}

__device__ longlong1 operator+(longlong1 a, longlong1 b) {
  // CHECK: return int64_t(a + b);
  return make_longlong1(a.x + b.x);
}

__global__ void test_make_longlong1() {
  typedef hipcub::BlockReduce<longlong1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: int64_t res = sycl::reduce_over_group({{.+}}, int64_t(1.), {{.+}});
  longlong1 res = BlockReduce(smem_storage).Sum(make_longlong1(1.));
}

__device__ longlong2 operator+(longlong2 a, longlong2 b) {
  // CHECK: return sycl::vec<long long, 2>(a.x() + b.x(), a.y() + b.y());
  return make_longlong2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_longlong2() {
  typedef hipcub::BlockReduce<longlong2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<long long, 2> res = sycl::reduce_over_group({{.+}}, sycl::vec<long long, 2>(1., 2.), {{.+}});
  longlong2 res = BlockReduce(smem_storage).Sum(make_longlong2(1., 2.));
}

__device__ longlong3 operator+(longlong3 a, longlong3 b) {
  // CHECK: return sycl::vec<long long, 3>(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_longlong3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_longlong3() {
  typedef hipcub::BlockReduce<longlong3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<long long, 3> res = sycl::reduce_over_group({{.+}}, sycl::vec<long long, 3>(1., 2., 3.), {{.+}});
  longlong3 res = BlockReduce(smem_storage).Sum(make_longlong3(1., 2., 3.));
}

__device__ longlong4 operator+(longlong4 a, longlong4 b) {
  // CHECK: return sycl::vec<long long, 4>(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_longlong4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_longlong4() {
  typedef hipcub::BlockReduce<longlong4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<long long, 4> res = sycl::reduce_over_group({{.+}}, sycl::vec<long long, 4>(1., 2., 3., 4.), {{.+}});
  longlong4 res = BlockReduce(smem_storage).Sum(make_longlong4(1., 2., 3., 4.));
}

__device__ ulonglong1 operator+(ulonglong1 a, ulonglong1 b) {
  // CHECK: return uint64_t(a + b);
  return make_ulonglong1(a.x + b.x);
}

__global__ void test_make_ulonglong1() {
  typedef hipcub::BlockReduce<ulonglong1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: uint64_t res = sycl::reduce_over_group({{.+}}, uint64_t(1.), {{.+}});
  ulonglong1 res = BlockReduce(smem_storage).Sum(make_ulonglong1(1.));
}

__device__ ulonglong2 operator+(ulonglong2 a, ulonglong2 b) {
  // CHECK: return sycl::vec<unsigned long long, 2>(a.x() + b.x(), a.y() + b.y());
  return make_ulonglong2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_ulonglong2() {
  typedef hipcub::BlockReduce<ulonglong2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<unsigned long long, 2> res = sycl::reduce_over_group({{.+}}, sycl::vec<unsigned long long, 2>(1., 2.), {{.+}});
  ulonglong2 res = BlockReduce(smem_storage).Sum(make_ulonglong2(1., 2.));
}

__device__ ulonglong3 operator+(ulonglong3 a, ulonglong3 b) {
  // CHECK: return sycl::vec<unsigned long long, 3>(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_ulonglong3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_ulonglong3() {
  typedef hipcub::BlockReduce<ulonglong3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<unsigned long long, 3> res = sycl::reduce_over_group({{.+}}, sycl::vec<unsigned long long, 3>(1., 2., 3.), {{.+}});
  ulonglong3 res = BlockReduce(smem_storage).Sum(make_ulonglong3(1., 2., 3.));
}

__device__ ulonglong4 operator+(ulonglong4 a, ulonglong4 b) {
  // CHECK: return sycl::vec<unsigned long long, 4>(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_ulonglong4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_ulonglong4() {
  typedef hipcub::BlockReduce<ulonglong4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::vec<unsigned long long, 4> res = sycl::reduce_over_group({{.+}}, sycl::vec<unsigned long long, 4>(1., 2., 3., 4.), {{.+}});
  ulonglong4 res = BlockReduce(smem_storage).Sum(make_ulonglong4(1., 2., 3., 4.));
}

__device__ float1 operator+(float1 a, float1 b) {
  // CHECK: return float(a + b);
  return make_float1(a.x + b.x);
}

__global__ void test_make_float1() {
  typedef hipcub::BlockReduce<float1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: float res = sycl::reduce_over_group({{.+}}, float(1.), {{.+}});
  float1 res = BlockReduce(smem_storage).Sum(make_float1(1.));
}

__device__ float2 operator+(float2 a, float2 b) {
  // CHECK: return sycl::float2(a.x() + b.x(), a.y() + b.y());
  return make_float2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_float2() {
  typedef hipcub::BlockReduce<float2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::float2 res = sycl::reduce_over_group({{.+}}, sycl::float2(1., 2.), {{.+}});
  float2 res = BlockReduce(smem_storage).Sum(make_float2(1., 2.));
}

__device__ float3 operator+(float3 a, float3 b) {
  // CHECK: return sycl::float3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_float3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_float3() {
  typedef hipcub::BlockReduce<float3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::float3 res = sycl::reduce_over_group({{.+}}, sycl::float3(1., 2., 3.), {{.+}});
  float3 res = BlockReduce(smem_storage).Sum(make_float3(1., 2., 3.));
}

__device__ float4 operator+(float4 a, float4 b) {
  // CHECK: return sycl::float4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_float4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_float4() {
  typedef hipcub::BlockReduce<float4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::float4 res = sycl::reduce_over_group({{.+}}, sycl::float4(1., 2., 3., 4.), {{.+}});
  float4 res = BlockReduce(smem_storage).Sum(make_float4(1., 2., 3., 4.));
}

__device__ double1 operator+(double1 a, double1 b) {
  // CHECK: return double(a + b);
  return make_double1(a.x + b.x);
}

__global__ void test_make_double1() {
  typedef hipcub::BlockReduce<double1, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: double res = sycl::reduce_over_group({{.+}}, double(1.), {{.+}});
  double1 res = BlockReduce(smem_storage).Sum(make_double1(1.));
}

__device__ double2 operator+(double2 a, double2 b) {
  // CHECK: return sycl::double2(a.x() + b.x(), a.y() + b.y());
  return make_double2(a.x + b.x, a.y + b.y);
}

__global__ void test_make_double2() {
  typedef hipcub::BlockReduce<double2, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::double2 res = sycl::reduce_over_group({{.+}}, sycl::double2(1., 2.), {{.+}});
  double2 res = BlockReduce(smem_storage).Sum(make_double2(1., 2.));
}

__device__ double3 operator+(double3 a, double3 b) {
  // CHECK: return sycl::double3(a.x() + b.x(), a.y() + b.y(), a.z() + b.z());
  return make_double3(a.x + b.x, a.y + b.y, a.z + b.z);
}

__global__ void test_make_double3() {
  typedef hipcub::BlockReduce<double3, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::double3 res = sycl::reduce_over_group({{.+}}, sycl::double3(1., 2., 3.), {{.+}});
  double3 res = BlockReduce(smem_storage).Sum(make_double3(1., 2., 3.));
}

__device__ double4 operator+(double4 a, double4 b) {
  // CHECK: return sycl::double4(a.x() + b.x(), a.y() + b.y(), a.z() + b.z(), a.w() + b.w());
  return make_double4(a.x + b.x, a.y + b.y, a.z + b.z, a.w + b.w);
}

__global__ void test_make_double4() {
  typedef hipcub::BlockReduce<double4, 128> BlockReduce;
  __shared__ typename BlockReduce::TempStorage smem_storage;
  // CHECK: sycl::double4 res = sycl::reduce_over_group({{.+}}, sycl::double4(1., 2., 3., 4.), {{.+}});
  double4 res = BlockReduce(smem_storage).Sum(make_double4(1., 2., 3., 4.));
}
#endif
