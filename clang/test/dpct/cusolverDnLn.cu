// RUN: dpct --format-range=none --report-format=csv --usm-level=none -out-root %T/cusolverDnLn %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn/cusolverDnLn.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusolverDnLn/cusolverDnLn.dp.cpp -o %T/cusolverDnLn/cusolverDnLn.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    //CHECK: *(&Lwork) = oneapi::mkl::lapack::potrf_scratchpad_size<float>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::potrf_scratchpad_size<double>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::potrf_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::potrf_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda);
    hipsolverDnSpotrf_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    hipsolverDnDpotrf_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    hipsolverDnCpotrf_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    hipsolverDnZpotrf_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    //CHECK: *(&Lwork) = oneapi::mkl::lapack::getrf_scratchpad_size<float>(**cusolverH, m, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::getrf_scratchpad_size<double>(**cusolverH, m, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<float>>(**cusolverH, m, n, lda);
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<double>>(**cusolverH, m, n, lda);
    hipsolverDnSgetrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnDgetrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnCgetrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnZgetrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);

    //CHECK: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);


    //CHECK: {
    //CHECK-NEXT: auto C_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&C_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<float>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<float, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_f_buf_ct{{[0-9]+}}, lda, B_f_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto C_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&C_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<float>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<float, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_f_buf_ct{{[0-9]+}}, lda, B_f_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: }
    status = hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&C_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<double>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<double, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto C_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&C_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<double>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<double, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_d_buf_ct{{[0-9]+}}, lda, B_d_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: }
    status = hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&C_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<float>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<float>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&C_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<float>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<float>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: }
    status = hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);


    //CHECK: {
    //CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&C_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<double>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<double>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&C_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<double>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<double>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, C_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: }
    status = hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<float>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnSgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_f_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<float>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnSgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_f_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<double>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnDgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_d_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<double>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnDgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_d_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<float>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnCgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_c_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<float>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnCgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_c_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<double>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnZgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_z_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<double>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnZgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, workspace_z_buf_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<float>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<float, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_f_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<float>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<float, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_f_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<double>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<double, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_d_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<double>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<double, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_d_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<float>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<float>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_c_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<float>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<float>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_c_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<double>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<double>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_z_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}}));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer6(sycl::range<1>(1));
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<double>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: sycl::buffer<std::complex<double>, 1> scratchpad_ct{{[0-9]+}}{sycl::range<1>(scratchpad_size_ct{{[0-9]+}})};
    //CHECK-NEXT: oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer6, B_z_buf_ct{{[0-9]+}}, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer6.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<float>(**cusolverH, m, n, lda));
    //CHECK-NEXT: Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<float>(**cusolverH, m, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    hipsolverDnSgeqrf_bufferSize(*cusolverH, m, n, &A_f, lda, &Lwork);
    status = hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<double>(**cusolverH, m, n, lda));
    //CHECK-NEXT: Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<double>(**cusolverH, m, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    hipsolverDnDgeqrf_bufferSize(*cusolverH, m, n, &A_d, lda, &Lwork);
    status = hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<std::complex<float>>(**cusolverH, m, n, lda));
    //CHECK-NEXT: Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<std::complex<float>>(**cusolverH, m, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    hipsolverDnCgeqrf_bufferSize(*cusolverH, m, n, &A_c, lda, &Lwork);
    status = hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<std::complex<double>>(**cusolverH, m, n, lda));
    //CHECK-NEXT: Lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<std::complex<double>>(**cusolverH, m, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    hipsolverDnZgeqrf_bufferSize(*cusolverH, m, n, &A_z, lda, &Lwork);
    status = hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ormqr_scratchpad_size<float>(**cusolverH, side, trans, m, n, k, lda, ldc));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ormqr_scratchpad_size<float>(**cusolverH, side, trans, m, n, k, lda, ldc);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, B_f_buf_ct{{[0-9]+}}, ldb, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, B_f_buf_ct{{[0-9]+}}, ldb, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    hipsolverDnSormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &C_f, ldc, &Lwork);
    status = hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ormqr_scratchpad_size<double>(**cusolverH, side, trans, m, n, k, lda, ldc));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ormqr_scratchpad_size<double>(**cusolverH, side, trans, m, n, k, lda, ldc);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, B_d_buf_ct{{[0-9]+}}, ldb, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, B_d_buf_ct{{[0-9]+}}, ldb, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    hipsolverDnDormqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &C_d, ldc, &Lwork);
    status = hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);


    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::unmqr_scratchpad_size<std::complex<float>>(**cusolverH, side, trans, m, n, k, lda, ldc));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::unmqr_scratchpad_size<std::complex<float>>(**cusolverH, side, trans, m, n, k, lda, ldc);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, B_c_buf_ct{{[0-9]+}}, ldb, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, B_c_buf_ct{{[0-9]+}}, ldb, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    hipsolverDnCunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &C_c, ldc, &Lwork);
    status = hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::unmqr_scratchpad_size<std::complex<double>>(**cusolverH, side, trans, m, n, k, lda, ldc));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::unmqr_scratchpad_size<std::complex<double>>(**cusolverH, side, trans, m, n, k, lda, ldc);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, B_z_buf_ct{{[0-9]+}}, ldb, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, B_z_buf_ct{{[0-9]+}}, ldb, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    hipsolverDnZunmqr_bufferSize(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &C_z, ldc, &Lwork);
    status = hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgqr_scratchpad_size<float>(**cusolverH, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgqr_scratchpad_size<float>(**cusolverH, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgqr_bufferSize(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgqr_scratchpad_size<double>(**cusolverH, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgqr_scratchpad_size<double>(**cusolverH, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgqr_bufferSize(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungqr_scratchpad_size<std::complex<float>>(**cusolverH, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungqr_scratchpad_size<std::complex<float>>(**cusolverH, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungqr_bufferSize(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungqr_scratchpad_size<std::complex<double>>(**cusolverH, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungqr_scratchpad_size<std::complex<double>>(**cusolverH, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungqr_bufferSize(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<float>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<float>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    hipsolverDnSsytrf_bufferSize(*cusolverH, n, &A_f, lda, &Lwork);
    status = hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<double>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<double>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    hipsolverDnDsytrf_bufferSize(*cusolverH, n, &A_d, lda, &Lwork);
    status = hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<std::complex<float>>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<std::complex<float>>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    hipsolverDnCsytrf_bufferSize(*cusolverH, n, &A_c, lda, &Lwork);
    status = hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<std::complex<double>>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::uplo uplo_ct_mkl_upper_lower;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrf_scratchpad_size<std::complex<double>>(**cusolverH, uplo_ct_mkl_upper_lower, n, lda);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto devIpiv_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(&devIpiv);
    //CHECK-NEXT: sycl::buffer<int64_t> result_temp_buffer5(sycl::range<1>(1));
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, result_temp_buffer5, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: devIpiv_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)result_temp_buffer5.get_access<sycl::access_mode::read>()[0];
    //CHECK-NEXT: }
    status = hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    hipsolverDnZsytrf_bufferSize(*cusolverH, n, &A_z, lda, &Lwork);
    status = hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}

