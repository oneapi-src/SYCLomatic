// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5
// RUN: dpct --format-range=none -out-root %T/math/bfloat16/bfloat16 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/bfloat16/bfloat16/bfloat16.dp.cpp


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

// CHECK: class C : public sycl::marray<sycl::ext::oneapi::bfloat16, 2> {
class C : public __hip_bfloat162 {
  void f() {
    // CHECK: (*this)[0];
    // CHECK-NEXT: (*this)[1];
    x;
    y;
  }
};

// CHECK: void foo(sycl::ext::oneapi::bfloat16 *a, sycl::marray<sycl::ext::oneapi::bfloat16, 2> *b) {
void foo(__hip_bfloat16 *a, __hip_bfloat162 *b) {
  int i = 0;
  float f = 3.0f;
  // CHECK: a[i] = (sycl::ext::oneapi::bfloat16)f;
  a[i] = (__hip_bfloat16)f;

  // CHECK: (*b)[0];
  // CHECK-NEXT: (*b)[1];
  b->x;
  b->y;
}

__global__ void kernelFuncBfloat16Arithmetic() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  __hip_bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  // CHECK: bf16 = sycl::fabs(float(bf16_1));
  bf16 = __habs(bf16_1);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 + bf16_2, 0.f, 1.0f);
  bf16 = __hadd_sat(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 / bf16_2;
  bf16 = __hdiv(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2 + bf16_3;
  bf16 = __hfma(bf16_1, bf16_2, bf16_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma_relu is not supported.
  // CHECK-NEXT: */
  bf16 = __hfma_relu(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2 + bf16_3, 0.f, 1.0f);
  bf16 = __hfma_sat(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2, 0.f, 1.0f);
  bf16 = __hmul_sat(bf16_1, bf16_2);
  // CHECK: bf16 = -bf16_1;
  bf16 = __hneg(bf16_1);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 - bf16_2, 0.f, 1.0f);
  bf16 = __hsub_sat(bf16_1, bf16_2);
}

__global__ void kernelFuncBfloat162Arithmetic() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2, bf162_3;
  __hip_bfloat162 bf162, bf162_1, bf162_2, bf162_3;
  // CHECK: bf162 = bf162_1 / bf162_2;
  bf162 = __h2div(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::fabs(float(bf162_1[0])), sycl::fabs(float(bf162_1[1])));
  bf162 = __habs2(bf162_1);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 + bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hadd2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::complex_mul_add(bf162_1, bf162_2, bf162_3);
  bf162 = __hcmadd(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2 + bf162_3;
  bf162 = __hfma2(bf162_1, bf162_2, bf162_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma2_relu is not supported.
  // CHECK-NEXT: */
  bf162 = __hfma2_relu(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2 + bf162_3, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hfma2_sat(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hmul2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = -bf162_1;
  bf162 = __hneg2(bf162_1);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 - bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hsub2_sat(bf162_1, bf162_2);
}

// CHECK: void test_conversions_device(sycl::ext::oneapi::bfloat16 *deviceArrayBFloat16) {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
__global__ void test_conversions_device(__hip_bfloat16 *deviceArrayBFloat16) {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);

  // CHECK:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  bf16_2 = __ldca(deviceArrayBFloat16);
  bf16_2 = __ldca(&bf16);
  bf162_2 = __ldca(&bf162);
  bf16_2 = __ldcg(deviceArrayBFloat16);
  bf16_2 = __ldcg(&bf16);
  bf162_2 = __ldcg(&bf162);
  bf16_2 = __ldcs(deviceArrayBFloat16);
  bf16_2 = __ldcs(&bf16);
  bf162_2 = __ldcs(&bf162);
  bf16_2 = __ldcv(deviceArrayBFloat16);
  bf16_2 = __ldcv(&bf16);
  bf162_2 = __ldcv(&bf162);
  bf16_2 = __ldg(deviceArrayBFloat16);
  bf16_2 = __ldg(&bf16);
  bf162_2 = __ldg(&bf162);
  bf16_2 = __ldlu(deviceArrayBFloat16);
  bf16_2 = __ldlu(&bf16);
  bf162_2 = __ldlu(&bf162);

  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  __stcg(deviceArrayBFloat16 + 1, bf16);
  __stcg(&bf16_2, bf16);
  __stcg(&bf162_2, bf162);
  __stcs(deviceArrayBFloat16, bf16);
  __stcs(&bf16_2, bf16);
  __stcs(&bf162_2, bf162);
  __stwb(deviceArrayBFloat16 + 1, bf16);
  __stwb(&bf16_2, bf16);
  __stwb(&bf162_2, bf162);
  __stwt(deviceArrayBFloat16, bf16);
  __stwt(&bf16_2, bf16);
  __stwt(&bf162_2, bf162);
}

// CHECK: void test_conversions() {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
void test_conversions() {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);
}

int main() { return 0; }
