// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5
// RUN: dpct --format-range=none -out-root %T/math/bfloat16/bfloat16 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/math/bfloat16/bfloat16/bfloat16.dp.cpp


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"

// CHECK: class C : public sycl::marray<sycl::ext::oneapi::bfloat16, 2> {
class C : public __hip_bfloat162 {
  void f() {
    // CHECK: (*this)[0];
    // CHECK-NEXT: (*this)[1];
    x;
    y;
  }
};

// CHECK: void foo(sycl::ext::oneapi::bfloat16 *a, sycl::marray<sycl::ext::oneapi::bfloat16, 2> *b) {
void foo(__hip_bfloat16 *a, __hip_bfloat162 *b) {
  int i = 0;
  float f = 3.0f;
  // CHECK: a[i] = (sycl::ext::oneapi::bfloat16)f;
  a[i] = (__hip_bfloat16)f;

  // CHECK: (*b)[0];
  // CHECK-NEXT: (*b)[1];
  b->x;
  b->y;
}

__global__ void kernelFuncBfloat16Arithmetic() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  __hip_bfloat16 bf16, bf16_1, bf16_2, bf16_3;
  // CHECK: bf16 = sycl::fabs(float(bf16_1));
  bf16 = __habs(bf16_1);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 + bf16_2;
  bf16 = __hadd_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 + bf16_2, 0.f, 1.0f);
  bf16 = __hadd_sat(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 / bf16_2;
  bf16 = __hdiv(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2 + bf16_3;
  bf16 = __hfma(bf16_1, bf16_2, bf16_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma_relu is not supported.
  // CHECK-NEXT: */
  bf16 = __hfma_relu(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2 + bf16_3, 0.f, 1.0f);
  bf16 = __hfma_sat(bf16_1, bf16_2, bf16_3);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 * bf16_2;
  bf16 = __hmul_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 * bf16_2, 0.f, 1.0f);
  bf16 = __hmul_sat(bf16_1, bf16_2);
  // CHECK: bf16 = -bf16_1;
  bf16 = __hneg(bf16_1);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub(bf16_1, bf16_2);
  // CHECK: bf16 = bf16_1 - bf16_2;
  bf16 = __hsub_rn(bf16_1, bf16_2);
  // CHECK: bf16 = dpct::clamp<sycl::ext::oneapi::bfloat16>(bf16_1 - bf16_2, 0.f, 1.0f);
  bf16 = __hsub_sat(bf16_1, bf16_2);
}

__global__ void kernelFuncBfloat162Arithmetic() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2, bf162_3;
  __hip_bfloat162 bf162, bf162_1, bf162_2, bf162_3;
  // CHECK: bf162 = bf162_1 / bf162_2;
  bf162 = __h2div(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::fabs(float(bf162_1[0])), sycl::fabs(float(bf162_1[1])));
  bf162 = __habs2(bf162_1);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 + bf162_2;
  bf162 = __hadd2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 + bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hadd2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::complex_mul_add(bf162_1, bf162_2, bf162_3);
  bf162 = __hcmadd(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2 + bf162_3;
  bf162 = __hfma2(bf162_1, bf162_2, bf162_3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma2_relu is not supported.
  // CHECK-NEXT: */
  bf162 = __hfma2_relu(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2 + bf162_3, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hfma2_sat(bf162_1, bf162_2, bf162_3);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 * bf162_2;
  bf162 = __hmul2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 * bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hmul2_sat(bf162_1, bf162_2);
  // CHECK: bf162 = -bf162_1;
  bf162 = __hneg2(bf162_1);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2(bf162_1, bf162_2);
  // CHECK: bf162 = bf162_1 - bf162_2;
  bf162 = __hsub2_rn(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::clamp(bf162_1 - bf162_2, {0.f, 0.f}, {1.f, 1.f});
  bf162 = __hsub2_sat(bf162_1, bf162_2);
}

__global__ void kernelFuncBfloat16Comparison() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16_1, bf16_2;
  __hip_bfloat16 bf16_1, bf16_2;
  bool b;
  // CHECK: b = bf16_1 == bf16_2;
  b = __heq(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::equal_to<>());
  b = __hequ(bf16_1, bf16_2);
  // CHECK: b = bf16_1 >= bf16_2;
  b = __hge(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::greater_equal<>());
  b = __hgeu(bf16_1, bf16_2);
  // CHECK: b = bf16_1 > bf16_2;
  b = __hgt(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::greater<>());
  b = __hgtu(bf16_1, bf16_2);
  // CHECK: b = sycl::isinf(float(bf16_1));
  b = __hisinf(bf16_1);
  // CHECK: b = sycl::isnan(float(bf16_1));
  b = __hisnan(bf16_1);
  // CHECK: b = bf16_1 <= bf16_2;
  b = __hle(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::less_equal<>());
  b = __hleu(bf16_1, bf16_2);
  // CHECK: b = bf16_1 < bf16_2;
  b = __hlt(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::less<>());
  b = __hltu(bf16_1, bf16_2);
  // CHECK: b = sycl::fmax(float(bf16_1), float(bf16_2));
  b = __hmax(bf16_1, bf16_2);
  // CHECK: b = dpct::fmax_nan(bf16_1, bf16_2);
  b = __hmax_nan(bf16_1, bf16_2);
  // CHECK: b = sycl::fmin(float(bf16_1), float(bf16_2));
  b = __hmin(bf16_1, bf16_2);
  // CHECK: b = dpct::fmin_nan(bf16_1, bf16_2);
  b = __hmin_nan(bf16_1, bf16_2);
  // CHECK: b = dpct::compare(bf16_1, bf16_2, std::not_equal_to<>());
  b = __hne(bf16_1, bf16_2);
  // CHECK: b = dpct::unordered_compare(bf16_1, bf16_2, std::not_equal_to<>());
  b = __hneu(bf16_1, bf16_2);
}

__global__ void kernelFuncBfloat162Comparison() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  bool b;
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::equal_to<>());
  b = __hbeq2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::equal_to<>());
  b = __hbequ2(bf162_1, bf162_2);
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::greater_equal<>());
  b = __hbge2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::greater_equal<>());
  b = __hbgeu2(bf162_1, bf162_2);
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::greater<>());
  b = __hbgt2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::greater<>());
  b = __hbgtu2(bf162_1, bf162_2);
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::less_equal<>());
  b = __hble2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::less_equal<>());
  b = __hbleu2(bf162_1, bf162_2);
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::less<>());
  b = __hblt2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::less<>());
  b = __hbltu2(bf162_1, bf162_2);
  // CHECK: b = dpct::compare_both(bf162_1, bf162_2, std::not_equal_to<>());
  b = __hbne2(bf162_1, bf162_2);
  // CHECK: b = dpct::unordered_compare_both(bf162_1, bf162_2, std::not_equal_to<>());
  b = __hbneu2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::equal_to<>());
  bf162 = __heq2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::equal_to<>());
  bf162 = __hequ2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::greater_equal<>());
  bf162 = __hge2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::greater_equal<>());
  bf162 = __hgeu2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::greater<>());
  bf162 = __hgt2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::greater<>());
  bf162 = __hgtu2(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::isnan(float(bf162_1[0])), sycl::isnan(float(bf162_1[1])));
  bf162 = __hisnan2(bf162_1);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::less_equal<>());
  bf162 = __hle2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::less_equal<>());
  bf162 = __hleu2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::less<>());
  bf162 = __hlt2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::less<>());
  bf162 = __hltu2(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::fmax(float(bf162_1[0]), float(bf162_2[0])), sycl::fmax(float(bf162_1[1]), float(bf162_2[1])));
  bf162 = __hmax2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::fmax_nan(bf162_1, bf162_2);
  bf162 = __hmax2_nan(bf162_1, bf162_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::fmin(float(bf162_1[0]), float(bf162_2[0])), sycl::fmin(float(bf162_1[1]), float(bf162_2[1])));
  bf162 = __hmin2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::fmin_nan(bf162_1, bf162_2);
  bf162 = __hmin2_nan(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::compare(bf162_1, bf162_2, std::not_equal_to<>());
  bf162 = __hne2(bf162_1, bf162_2);
  // CHECK: bf162 = dpct::unordered_compare(bf162_1, bf162_2, std::not_equal_to<>());
  bf162 = __hneu2(bf162_1, bf162_2);
}

// CHECK: void test_conversions_device(sycl::ext::oneapi::bfloat16 *deviceArrayBFloat16) {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
__global__ void test_conversions_device(__hip_bfloat16 *deviceArrayBFloat16) {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  int i;
  long long ll;
  short s;
  unsigned u;
  unsigned long long ull;
  unsigned short us;
  double d;
  // CHECK: f2 = sycl::float2(bf162[0], bf162[1]);
  f2 = __bfloat1622float2(bf162);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf16, bf16);
  bf162 = __bfloat162bfloat162(bf16);
  // CHECK: f = static_cast<float>(bf16);
  f = __bfloat162float(bf16);
  // CHECK: i = sycl::vec<float, 1>(bf16).convert<int, sycl::rounding_mode::rtn>()[0];
  i = __bfloat162int_rd(bf16);
  // CHECK: i = sycl::vec<float, 1>(bf16).convert<int, sycl::rounding_mode::rte>()[0];
  i = __bfloat162int_rn(bf16);
  // CHECK: i = sycl::vec<float, 1>(bf16).convert<int, sycl::rounding_mode::rtp>()[0];
  i = __bfloat162int_ru(bf16);
  // CHECK: i = sycl::vec<float, 1>(bf16).convert<int, sycl::rounding_mode::rtz>()[0];
  i = __bfloat162int_rz(bf16);
  // CHECK: ll = sycl::vec<float, 1>(bf16).convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __bfloat162ll_rd(bf16);
  // CHECK: ll = sycl::vec<float, 1>(bf16).convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __bfloat162ll_rn(bf16);
  // CHECK: ll = sycl::vec<float, 1>(bf16).convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __bfloat162ll_ru(bf16);
  // CHECK: ll = sycl::vec<float, 1>(bf16).convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __bfloat162ll_rz(bf16);
  // CHECK: s = sycl::vec<float, 1>(bf16).convert<short, sycl::rounding_mode::rtn>()[0];
  s = __bfloat162short_rd(bf16);
  // CHECK: s = sycl::vec<float, 1>(bf16).convert<short, sycl::rounding_mode::rte>()[0];
  s = __bfloat162short_rn(bf16);
  // CHECK: s = sycl::vec<float, 1>(bf16).convert<short, sycl::rounding_mode::rtp>()[0];
  s = __bfloat162short_ru(bf16);
  // CHECK: s = sycl::vec<float, 1>(bf16).convert<short, sycl::rounding_mode::rtz>()[0];
  s = __bfloat162short_rz(bf16);
  // CHECK: u = sycl::vec<float, 1>(bf16).convert<unsigned, sycl::rounding_mode::rtn>()[0];
  u = __bfloat162uint_rd(bf16);
  // CHECK: u = sycl::vec<float, 1>(bf16).convert<unsigned, sycl::rounding_mode::rte>()[0];
  u = __bfloat162uint_rn(bf16);
  // CHECK: u = sycl::vec<float, 1>(bf16).convert<unsigned, sycl::rounding_mode::rtp>()[0];
  u = __bfloat162uint_ru(bf16);
  // CHECK: u = sycl::vec<float, 1>(bf16).convert<unsigned, sycl::rounding_mode::rtz>()[0];
  u = __bfloat162uint_rz(bf16);
  // CHECK: ull = sycl::vec<float, 1>(bf16).convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __bfloat162ull_rd(bf16);
  // CHECK: ull = sycl::vec<float, 1>(bf16).convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __bfloat162ull_rn(bf16);
  // CHECK: ull = sycl::vec<float, 1>(bf16).convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __bfloat162ull_ru(bf16);
  // CHECK: ull = sycl::vec<float, 1>(bf16).convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __bfloat162ull_rz(bf16);
  // CHECK: us = sycl::vec<float, 1>(bf16).convert<unsigned short, sycl::rounding_mode::rtn>()[0];
  us = __bfloat162ushort_rd(bf16);
  // CHECK: us = sycl::vec<float, 1>(bf16).convert<unsigned short, sycl::rounding_mode::rte>()[0];
  us = __bfloat162ushort_rn(bf16);
  // CHECK: us = sycl::vec<float, 1>(bf16).convert<unsigned short, sycl::rounding_mode::rtp>()[0];
  us = __bfloat162ushort_ru(bf16);
  // CHECK: us = sycl::vec<float, 1>(bf16).convert<unsigned short, sycl::rounding_mode::rtz>()[0];
  us = __bfloat162ushort_rz(bf16);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __bfloat16_as_short is not supported.
  // CHECK-NEXT: */
  s = __bfloat16_as_short(bf16);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __bfloat16_as_ushort is not supported.
  // CHECK-NEXT: */
  us = __bfloat16_as_ushort(bf16);
  // CHECK: bf16 = sycl::ext::oneapi::bfloat16(d);
  bf16 = __double2bfloat16(d);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
  bf162 = __float22bfloat162_rn(f2);
  // CHECK: bf16 = sycl::ext::oneapi::bfloat16(f);
  bf16 = __float2bfloat16(f);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f, f);
  bf162 = __float2bfloat162_rn(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __float2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __float2bfloat16_rd(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __float2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __float2bfloat16_rn(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __float2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __float2bfloat16_ru(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __float2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __float2bfloat16_rz(f);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f_1, f_2);
  bf162 = __floats2bfloat162_rn(f_1, f_2);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf16_1, bf16_2);
  bf162 = __halves2bfloat162(bf16_1, bf16_2);
  // CHECK: bf16 = sycl::ext::oneapi::bfloat16(bf162[1]);
  bf16 = __high2bfloat16(bf162);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf162[1], bf162[1]);
  bf162 = __high2bfloat162(bf162);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf162_1[1], bf162_2[1]);
  bf162 = __highs2bfloat162(bf162_1, bf162_2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __int2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __int2bfloat16_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __int2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __int2bfloat16_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __int2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __int2bfloat16_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __int2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __int2bfloat16_rz(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ll2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __ll2bfloat16_rd(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ll2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __ll2bfloat16_rn(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ll2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __ll2bfloat16_ru(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ll2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __ll2bfloat16_rz(ll);
  // CHECK: bf16 = sycl::ext::oneapi::bfloat16(bf162[0]);
  bf16 = __low2bfloat16(bf162);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf162[0], bf162[0]);
  bf162 = __low2bfloat162(bf162);
  // CHECK: bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(bf162_1[0], bf162_2[0]);
  bf162 = __lows2bfloat162(bf162_1, bf162_2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __short2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __short2bfloat16_rd(s);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __short2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __short2bfloat16_rn(s);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __short2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __short2bfloat16_ru(s);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __short2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __short2bfloat16_rz(s);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __short_as_bfloat16 is not supported.
  // CHECK-NEXT: */
  bf16 = __short_as_bfloat16(s);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __uint2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __uint2bfloat16_rd(u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __uint2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __uint2bfloat16_rn(u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __uint2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __uint2bfloat16_ru(u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __uint2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __uint2bfloat16_rz(u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ull2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __ull2bfloat16_rd(ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ull2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __ull2bfloat16_rn(ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ull2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __ull2bfloat16_ru(ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ull2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __ull2bfloat16_rz(ull);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ushort2bfloat16_rd is not supported.
  // CHECK-NEXT: */
  bf16 = __ushort2bfloat16_rd(us);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ushort2bfloat16_rn is not supported.
  // CHECK-NEXT: */
  bf16 = __ushort2bfloat16_rn(us);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ushort2bfloat16_ru is not supported.
  // CHECK-NEXT: */
  bf16 = __ushort2bfloat16_ru(us);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ushort2bfloat16_rz is not supported.
  // CHECK-NEXT: */
  bf16 = __ushort2bfloat16_rz(us);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __ushort_as_bfloat16 is not supported.
  // CHECK-NEXT: */
  bf16 = __ushort_as_bfloat16(us);

  // CHECK:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldca call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldcv call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = *deviceArrayBFloat16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf16_2 = bf16;
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1098:{{[0-9]+}}: The '*' expression is used instead of the __ldlu call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   bf162_2 = bf162;
  bf16_2 = __ldca(deviceArrayBFloat16);
  bf16_2 = __ldca(&bf16);
  bf162_2 = __ldca(&bf162);
  bf16_2 = __ldcg(deviceArrayBFloat16);
  bf16_2 = __ldcg(&bf16);
  bf162_2 = __ldcg(&bf162);
  bf16_2 = __ldcs(deviceArrayBFloat16);
  bf16_2 = __ldcs(&bf16);
  bf162_2 = __ldcs(&bf162);
  bf16_2 = __ldcv(deviceArrayBFloat16);
  bf16_2 = __ldcv(&bf16);
  bf162_2 = __ldcv(&bf162);
  bf16_2 = __ldg(deviceArrayBFloat16);
  bf16_2 = __ldg(&bf16);
  bf162_2 = __ldg(&bf162);
  bf16_2 = __ldlu(deviceArrayBFloat16);
  bf16_2 = __ldlu(&bf16);
  bf162_2 = __ldlu(&bf162);

  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcg call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stcs call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *(deviceArrayBFloat16 + 1) = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwb call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: *deviceArrayBFloat16 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf16_2 = bf16;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The '=' expression is used instead of the __stwt call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: bf162_2 = bf162;
  __stcg(deviceArrayBFloat16 + 1, bf16);
  __stcg(&bf16_2, bf16);
  __stcg(&bf162_2, bf162);
  __stcs(deviceArrayBFloat16, bf16);
  __stcs(&bf16_2, bf16);
  __stcs(&bf162_2, bf162);
  __stwb(deviceArrayBFloat16 + 1, bf16);
  __stwb(&bf16_2, bf16);
  __stwb(&bf162_2, bf162);
  __stwt(deviceArrayBFloat16, bf16);
  __stwt(&bf16_2, bf16);
  __stwt(&bf162_2, bf162);
}

// CHECK: void test_conversions() {
// CHECK-NEXT:   float f, f_1, f_2;
// CHECK-NEXT:   sycl::float2 f2, f2_1, f2_2;
// CHECK-NEXT:   sycl::ext::oneapi::bfloat16 bf16, bf16_1, bf16_2;
// CHECK-NEXT:   sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1, bf162_2;
// CHECK-NEXT:   f2 = sycl::float2(bf162[0], bf162[1]);
// CHECK-NEXT:   f = static_cast<float>(bf16);
// CHECK-NEXT:   bf162 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(f2[0], f2[1]);
// CHECK-NEXT:   bf16 = sycl::ext::oneapi::bfloat16(f);
void test_conversions() {
  float f, f_1, f_2;
  float2 f2, f2_1, f2_2;
  __hip_bfloat16 bf16, bf16_1, bf16_2;
  __hip_bfloat162 bf162, bf162_1, bf162_2;
  f2 = __bfloat1622float2(bf162);
  f = __bfloat162float(bf16);
  bf162 = __float22bfloat162_rn(f2);
  bf16 = __float2bfloat16(f);
}

__global__ void kernelFuncBfloat16Math() {
  // CHECK: sycl::ext::oneapi::bfloat16 bf16, bf16_1;
  __hip_bfloat16 bf16, bf16_1;
  // CHECK: bf16_1 = sycl::ceil(float(bf16));
  bf16_1 = hceil(bf16);
  // CHECK: bf16_1 = sycl::cos(float(bf16));
  bf16_1 = hcos(bf16);
  // CHECK: bf16_1 = sycl::exp(float(bf16));
  bf16_1 = hexp(bf16);
  // CHECK: bf16_1 = sycl::exp10(float(bf16));
  bf16_1 = hexp10(bf16);
  // CHECK: bf16_1 = sycl::exp2(float(bf16));
  bf16_1 = hexp2(bf16);
  // CHECK: bf16_1 = sycl::floor(float(bf16));
  bf16_1 = hfloor(bf16);
  // CHECK: bf16_1 = sycl::log(float(bf16));
  bf16_1 = hlog(bf16);
  // CHECK: bf16_1 = sycl::log10(float(bf16));
  bf16_1 = hlog10(bf16);
  // CHECK: bf16_1 = sycl::log2(float(bf16));
  bf16_1 = hlog2(bf16);
  // CHECK: bf16_1 = sycl::half_precision::recip(float(bf16));
  bf16_1 = hrcp(bf16);
  // CHECK: bf16_1 = sycl::rint(float(bf16));
  bf16_1 = hrint(bf16);
  // CHECK: bf16_1 = sycl::rsqrt(float(bf16));
  bf16_1 = hrsqrt(bf16);
  // CHECK: bf16_1 = sycl::sin(float(bf16));
  bf16_1 = hsin(bf16);
  // CHECK: bf16_1 = sycl::sqrt(float(bf16));
  bf16_1 = hsqrt(bf16);
  // CHECK: bf16_1 = sycl::trunc(float(bf16));
  bf16_1 = htrunc(bf16);
}

__global__ void kernelFuncBfloat162Math() {
  // CHECK: sycl::marray<sycl::ext::oneapi::bfloat16, 2> bf162, bf162_1;
  __hip_bfloat162 bf162, bf162_1;
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::ceil(float(bf162[0])), sycl::ceil(float(bf162[1])));
  bf162_1 = h2ceil(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::cos(float(bf162[0])), sycl::cos(float(bf162[1])));
  bf162_1 = h2cos(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp(float(bf162[0])), sycl::exp(float(bf162[1])));
  bf162_1 = h2exp(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp10(float(bf162[0])), sycl::exp10(float(bf162[1])));
  bf162_1 = h2exp10(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::exp2(float(bf162[0])), sycl::exp2(float(bf162[1])));
  bf162_1 = h2exp2(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::floor(float(bf162[0])), sycl::floor(float(bf162[1])));
  bf162_1 = h2floor(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log(float(bf162[0])), sycl::log(float(bf162[1])));
  bf162_1 = h2log(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log10(float(bf162[0])), sycl::log10(float(bf162[1])));
  bf162_1 = h2log10(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::log2(float(bf162[0])), sycl::log2(float(bf162[1])));
  bf162_1 = h2log2(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::half_precision::recip(float(bf162[0])), sycl::half_precision::recip(float(bf162[1])));
  bf162_1 = h2rcp(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::rint(float(bf162[0])), sycl::rint(float(bf162[1])));
  bf162_1 = h2rint(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::rsqrt(float(bf162[0])), sycl::rsqrt(float(bf162[1])));
  bf162_1 = h2rsqrt(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::sin(float(bf162[0])), sycl::sin(float(bf162[1])));
  bf162_1 = h2sin(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::sqrt(float(bf162[0])), sycl::sqrt(float(bf162[1])));
  bf162_1 = h2sqrt(bf162);
  // CHECK: bf162_1 = sycl::marray<sycl::ext::oneapi::bfloat16, 2>(sycl::trunc(float(bf162[0])), sycl::trunc(float(bf162[1])));
  bf162_1 = h2trunc(bf162);
}

int main() { return 0; }
