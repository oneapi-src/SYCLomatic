// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/math/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/math/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/ext/intel/math.hpp>

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;
  float f0;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i0((double)f0);
  d2 = cyl_bessel_i0(f0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i0(d0);
  d2 = cyl_bessel_i0(d0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i1((double)f0);
  d2 = cyl_bessel_i1(f0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i1(d0);
  d2 = cyl_bessel_i1(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfcinv(d0);
  d2 = erfcinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfinv(d0);
  d2 = erfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::j0((double)f0);
  d2 = j0(f0);
  // CHECK: d2 = sycl::ext::intel::math::j0(d0);
  d2 = j0(d0);
  // CHECK: d2 = sycl::ext::intel::math::j1((double)f0);
  d2 = j1(f0);
  // CHECK: d2 = sycl::ext::intel::math::j1(d0);
  d2 = j1(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorm(d0);
  d2 = normcdf(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorminv(d0);
  d2 = normcdfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(i, &d0);
  d2 = norm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(d1, &d0);
  d2 = norm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(i, &d0);
  d2 = rnorm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(d1, &d0);
  d2 = rnorm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::y0((double)f0);
  d2 = y0(f0);
  // CHECK: d2 = sycl::ext::intel::math::y0(d0);
  d2 = y0(d0);
  // CHECK: d2 = sycl::ext::intel::math::y1((double)f0);
  d2 = y1(f0);
  // CHECK: d2 = sycl::ext::intel::math::y1(d0);
  d2 = y1(d0);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;
  double d0;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i0(f0);
  f2 = cyl_bessel_i0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i0((float)d0);
  f2 = cyl_bessel_i0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i1(f0);
  f2 = cyl_bessel_i1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i1((float)d0);
  f2 = cyl_bessel_i1f(d0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv(f0);
  f2 = erfcinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv((float)i);
  f2 = erfcinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::erfinv(f0);
  f2 = erfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfinv((float)i);
  f2 = erfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::j0(f0);
  f2 = j0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::j0((float)d0);
  f2 = j0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::j1(f0);
  f2 = j1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::j1((float)d0);
  f2 = j1f(d0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm(f0);
  f2 = normcdff(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm((float)i);
  f2 = normcdff(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv(f0);
  f2 = normcdfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv((float)i);
  f2 = normcdfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::norm(i, &f0);
  f2 = normf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::norm(f1, &f0);
  f2 = normf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(i, &f0);
  f2 = rnormf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(f1, &f0);
  f2 = rnormf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::y0(f0);
  f2 = y0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::y0((float)d0);
  f2 = y0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::y1(f0);
  f2 = y1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::y1((float)d0);
  f2 = y1f(d0);
}

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul(h, h_1);
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hneg(h_1);
  h_2 = __hneg(h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub(h, h_1);
  h_2 = __hsub(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hadd(h, h_1);
  h_2 = __hadd(h, h_1);

  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::heq(h, h_1);
  b = __heq(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hge(h, h_1);
  b = __hge(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgt(h, h_1);
  b = __hgt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hle(h, h_1);
  b = __hle(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hlt(h, h_1);
  b = __hlt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hne(h, h_1);
  b = __hne(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::h2div(h2, h2_1);
  h2_2 = __h2div(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2(h2, h2_1);
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2(h2, h2_1);
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hneg2(h2_1);
  h2_2 = __hneg2(h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2(h2, h2_1);
  h2_2 = __hsub2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);

  // CHECK: h2_2 = sycl::ext::intel::math::hisnan2(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hle2(h2, h2_1);
  h2_2 = __hle2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hleu2(h2, h2);
  __hleu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hlt2(h2, h2_1);
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hltu2(h2, h2);
  __hltu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hne2(h2, h2_1);
  h2_2 = __hne2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hneu2(h2, h2);
  __hneu2(h2, h2);
}

__global__ void kernelFuncInt() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  // CHECK: ll = sycl::max<long long>(ll, ll);
  // CHECK-NEXT: ll = sycl::min<long long>(ll, ll);
  // CHECK-NEXT: ull = sycl::max<unsigned long long>(ull, ull);
  // CHECK-NEXT: ull = sycl::min<unsigned long long>(ull, ull);
  // CHECK-NEXT: u = sycl::max<unsigned int>(u, u);
  // CHECK-NEXT: u = sycl::min<unsigned int>(u, u);
  ll = llmax(ll, ll);
  ll = llmin(ll, ll);
  ull = ullmax(ull, ull);
  ull = ullmin(ull, ull);
  u = umax(u, u);
  u = umin(u, u);
}

__global__ void kernelFuncSIMD() {
  unsigned int u, u_1, u_2;

  // CHECK: u_2 = sycl::ext::intel::math::vabs2(u);
  u_2 = __vabs2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabs4(u);
  u_2 = __vabs4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffs2(u, u_1);
  u_2 = __vabsdiffs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffs4(u, u_1);
  u_2 = __vabsdiffs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffu2(u, u_1);
  u_2 = __vabsdiffu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffu4(u, u_1);
  u_2 = __vabsdiffu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsss2(u);
  u_2 = __vabsss2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabsss4(u);
  u_2 = __vabsss4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vadd2(u, u_1);
  u_2 = __vadd2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vadd4(u, u_1);
  u_2 = __vadd4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddss2(u, u_1);
  u_2 = __vaddss2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddss4(u, u_1);
  u_2 = __vaddss4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddus2(u, u_1);
  u_2 = __vaddus2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddus4(u, u_1);
  u_2 = __vaddus4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgs2(u, u_1);
  u_2 = __vavgs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgs4(u, u_1);
  u_2 = __vavgs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgu2(u, u_1);
  u_2 = __vavgu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgu4(u, u_1);
  u_2 = __vavgu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpeq2(u, u_1);
  u_2 = __vcmpeq2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpeq4(u, u_1);
  u_2 = __vcmpeq4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpges2(u, u_1);
  u_2 = __vcmpges2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpges4(u, u_1);
  u_2 = __vcmpges4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgeu2(u, u_1);
  u_2 = __vcmpgeu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgeu4(u, u_1);
  u_2 = __vcmpgeu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgts2(u, u_1);
  u_2 = __vcmpgts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgts4(u, u_1);
  u_2 = __vcmpgts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgtu2(u, u_1);
  u_2 = __vcmpgtu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgtu4(u, u_1);
  u_2 = __vcmpgtu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmples2(u, u_1);
  u_2 = __vcmples2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmples4(u, u_1);
  u_2 = __vcmples4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpleu2(u, u_1);
  u_2 = __vcmpleu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpleu4(u, u_1);
  u_2 = __vcmpleu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmplts2(u, u_1);
  u_2 = __vcmplts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmplts4(u, u_1);
  u_2 = __vcmplts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpltu2(u, u_1);
  u_2 = __vcmpltu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpltu4(u, u_1);
  u_2 = __vcmpltu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpne2(u, u_1);
  u_2 = __vcmpne2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpne4(u, u_1);
  u_2 = __vcmpne4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vhaddu2(u, u_1);
  u_2 = __vhaddu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vhaddu4(u, u_1);
  u_2 = __vhaddu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxs2(u, u_1);
  u_2 = __vmaxs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxs4(u, u_1);
  u_2 = __vmaxs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxu2(u, u_1);
  u_2 = __vmaxu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxu4(u, u_1);
  u_2 = __vmaxu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmins2(u, u_1);
  u_2 = __vmins2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmins4(u, u_1);
  u_2 = __vmins4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vminu2(u, u_1);
  u_2 = __vminu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vminu4(u, u_1);
  u_2 = __vminu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vneg2(u);
  u_2 = __vneg2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vneg4(u);
  u_2 = __vneg4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vnegss2(u);
  u_2 = __vnegss2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vnegss4(u);
  u_2 = __vnegss4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vsads2(u, u_1);
  u_2 = __vsads2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsads4(u, u_1);
  u_2 = __vsads4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsadu2(u, u_1);
  u_2 = __vsadu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsadu4(u, u_1);
  u_2 = __vsadu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vseteq2(u, u_1);
  u_2 = __vseteq2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vseteq4(u, u_1);
  u_2 = __vseteq4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetges2(u, u_1);
  u_2 = __vsetges2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetges4(u, u_1);
  u_2 = __vsetges4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgeu2(u, u_1);
  u_2 = __vsetgeu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgeu4(u, u_1);
  u_2 = __vsetgeu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgts2(u, u_1);
  u_2 = __vsetgts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgts4(u, u_1);
  u_2 = __vsetgts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgtu2(u, u_1);
  u_2 = __vsetgtu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgtu4(u, u_1);
  u_2 = __vsetgtu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetles2(u, u_1);
  u_2 = __vsetles2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetles4(u, u_1);
  u_2 = __vsetles4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetleu2(u, u_1);
  u_2 = __vsetleu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetleu4(u, u_1);
  u_2 = __vsetleu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetlts2(u, u_1);
  u_2 = __vsetlts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetlts4(u, u_1);
  u_2 = __vsetlts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetltu2(u, u_1);
  u_2 = __vsetltu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetltu4(u, u_1);
  u_2 = __vsetltu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetne2(u, u_1);
  u_2 = __vsetne2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetne4(u, u_1);
  u_2 = __vsetne4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsub2(u, u_1);
  u_2 = __vsub2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsub4(u, u_1);
  u_2 = __vsub4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubss2(u, u_1);
  u_2 = __vsubss2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubss4(u, u_1);
  u_2 = __vsubss4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubus2(u, u_1);
  u_2 = __vsubus2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubus4(u, u_1);
  u_2 = __vsubus4(u, u_1);
}

int main() { return 0; }
