// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/math/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/math/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/math/cuda-math-extension/cuda-math-extension.dp.cpp -o %T/math/cuda-math-extension/cuda-math-extension.dp.o %}

// CHECK: #include <sycl/ext/intel/math.hpp>

#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;
  float f0;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i0((double)f0);
  d2 = cyl_bessel_i0(f0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i0(d0);
  d2 = cyl_bessel_i0(d0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i1((double)f0);
  d2 = cyl_bessel_i1(f0);
  // CHECK: d2 = sycl::ext::intel::math::cyl_bessel_i1(d0);
  d2 = cyl_bessel_i1(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfcinv(d0);
  d2 = erfcinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfinv(d0);
  d2 = erfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::j0((double)f0);
  d2 = j0(f0);
  // CHECK: d2 = sycl::ext::intel::math::j0(d0);
  d2 = j0(d0);
  // CHECK: d2 = sycl::ext::intel::math::j1((double)f0);
  d2 = j1(f0);
  // CHECK: d2 = sycl::ext::intel::math::j1(d0);
  d2 = j1(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorm(d0);
  d2 = normcdf(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorminv(d0);
  d2 = normcdfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(i, &d0);
  d2 = norm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(d1, &d0);
  d2 = norm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(i, &d0);
  d2 = rnorm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(d1, &d0);
  d2 = rnorm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::y0((double)f0);
  d2 = y0(f0);
  // CHECK: d2 = sycl::ext::intel::math::y0(d0);
  d2 = y0(d0);
  // CHECK: d2 = sycl::ext::intel::math::y1((double)f0);
  d2 = y1(f0);
  // CHECK: d2 = sycl::ext::intel::math::y1(d0);
  d2 = y1(d0);

  // Double Precision Intrinsics

  // CHECK: d2 = sycl::ext::intel::math::dadd_rd(d0, d1);
  d2 = __dadd_rd(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dadd_rn(d0, d1);
  d2 = __dadd_rn(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dadd_ru(d0, d1);
  d2 = __dadd_ru(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dadd_rz(d0, d1);
  d2 = __dadd_rz(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dmul_rd(d0, d1);
  d2 = __dmul_rd(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dmul_rn(d0, d1);
  d2 = __dmul_rn(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dmul_ru(d0, d1);
  d2 = __dmul_ru(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dmul_rz(d0, d1);
  d2 = __dmul_rz(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dsub_rd(d0, d1);
  d2 = __dsub_rd(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dsub_rn(d0, d1);
  d2 = __dsub_rn(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dsub_ru(d0, d1);
  d2 = __dsub_ru(d0, d1);
  // CHECK: d2 = sycl::ext::intel::math::dsub_rz(d0, d1);
  d2 = __dsub_rz(d0, d1);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;
  double d0;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i0(f0);
  f2 = cyl_bessel_i0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i0((float)d0);
  f2 = cyl_bessel_i0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i1(f0);
  f2 = cyl_bessel_i1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::cyl_bessel_i1((float)d0);
  f2 = cyl_bessel_i1f(d0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv(f0);
  f2 = erfcinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv((float)i);
  f2 = erfcinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::erfinv(f0);
  f2 = erfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfinv((float)i);
  f2 = erfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::j0(f0);
  f2 = j0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::j0((float)d0);
  f2 = j0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::j1(f0);
  f2 = j1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::j1((float)d0);
  f2 = j1f(d0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm(f0);
  f2 = normcdff(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm((float)i);
  f2 = normcdff(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv(f0);
  f2 = normcdfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv((float)i);
  f2 = normcdfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::norm(i, &f0);
  f2 = normf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::norm(f1, &f0);
  f2 = normf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(i, &f0);
  f2 = rnormf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(f1, &f0);
  f2 = rnormf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::y0(f0);
  f2 = y0f(f0);
  // CHECK: f2 = sycl::ext::intel::math::y0((float)d0);
  f2 = y0f(d0);
  // CHECK: f2 = sycl::ext::intel::math::y1(f0);
  f2 = y1f(f0);
  // CHECK: f2 = sycl::ext::intel::math::y1((float)d0);
  f2 = y1f(d0);

  // Single Precision Intrinsics

  // CHECK: f2 = sycl::ext::intel::math::fadd_rd(f0, f1);
  f2 = __fadd_rd(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fadd_rn(f0, f1);
  f2 = __fadd_rn(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fadd_ru(f0, f1);
  f2 = __fadd_ru(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fadd_rz(f0, f1);
  f2 = __fadd_rz(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fmul_rd(f0, f1);
  f2 = __fmul_rd(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fmul_rn(f0, f1);
  f2 = __fmul_rn(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fmul_ru(f0, f1);
  f2 = __fmul_ru(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fmul_rz(f0, f1);
  f2 = __fmul_rz(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fsub_rd(f0, f1);
  f2 = __fsub_rd(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fsub_rn(f0, f1);
  f2 = __fsub_rn(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fsub_ru(f0, f1);
  f2 = __fsub_ru(f0, f1);
  // CHECK: f2 = sycl::ext::intel::math::fsub_rz(f0, f1);
  f2 = __fsub_rz(f0, f1);
}

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul(h, h_1);
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hneg(h_1);
  h_2 = __hneg(h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub(h, h_1);
  h_2 = __hsub(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hadd(h, h_1);
  h_2 = __hadd(h, h_1);

  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::heq(h, h_1);
  b = __heq(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hge(h, h_1);
  b = __hge(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgt(h, h_1);
  b = __hgt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hisinf(h);
  b = __hisinf(h);
  // CHECK: b = sycl::ext::intel::math::hisnan(h);
  b = __hisnan(h);
  // CHECK: b = sycl::ext::intel::math::hle(h, h_1);
  b = __hle(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hlt(h, h_1);
  b = __hlt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hne(h, h_1);
  b = __hne(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);

  // Half Math Functions

  // CHECK: h_2 = sycl::ext::intel::math::ceil(h);
  h_2 = hceil(h);
  // CHECK: h_2 = sycl::ext::intel::math::cos(h);
  h_2 = hcos(h);
  // CHECK: h_2 = sycl::ext::intel::math::exp(h);
  h_2 = hexp(h);
  // CHECK: h_2 = sycl::ext::intel::math::exp10(h);
  h_2 = hexp10(h);
  // CHECK: h_2 = sycl::ext::intel::math::exp2(h);
  h_2 = hexp2(h);
  // CHECK: h_2 = sycl::ext::intel::math::floor(h);
  h_2 = hfloor(h);
  // CHECK: h_2 = sycl::ext::intel::math::log(h);
  h_2 = hlog(h);
  // CHECK: h_2 = sycl::ext::intel::math::log10(h);
  h_2 = hlog10(h);
  // CHECK: h_2 = sycl::ext::intel::math::log2(h);
  h_2 = hlog2(h);
  // CHECK: h_2 = sycl::ext::intel::math::inv(h);
  h_2 = hrcp(h);
  // CHECK: h_2 = sycl::ext::intel::math::rint(h);
  h_2 = hrint(h);
  // CHECK: h_2 = sycl::ext::intel::math::rsqrt(h);
  h_2 = hrsqrt(h);
  // CHECK: h_2 = sycl::ext::intel::math::sin(h);
  h_2 = hsin(h);
  // CHECK: h_2 = sycl::ext::intel::math::sqrt(h);
  h_2 = hsqrt(h);
  // CHECK: h_2 = sycl::ext::intel::math::trunc(h);
  h_2 = htrunc(h);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::h2div(h2, h2_1);
  h2_2 = __h2div(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2(h2, h2_1);
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2(h2, h2_1);
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hneg2(h2_1);
  h2_2 = __hneg2(h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2(h2, h2_1);
  h2_2 = __hsub2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);

  // CHECK: h2_2 = sycl::ext::intel::math::hisnan2(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hle2(h2, h2_1);
  h2_2 = __hle2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hleu2(h2, h2);
  __hleu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hlt2(h2, h2_1);
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hltu2(h2, h2);
  __hltu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hne2(h2, h2_1);
  h2_2 = __hne2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hneu2(h2, h2);
  __hneu2(h2, h2);

  // Half2 Math Functions

  // CHECK: h2_2 = sycl::ext::intel::math::ceil(h2);
  h2_2 = h2ceil(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::cos(h2);
  h2_2 = h2cos(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::exp(h2);
  h2_2 = h2exp(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::exp10(h2);
  h2_2 = h2exp10(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::exp2(h2);
  h2_2 = h2exp2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::floor(h2);
  h2_2 = h2floor(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::log(h2);
  h2_2 = h2log(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::log10(h2);
  h2_2 = h2log10(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::log2(h2);
  h2_2 = h2log2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::inv(h2);
  h2_2 = h2rcp(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::rint(h2);
  h2_2 = h2rint(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::rsqrt(h2);
  h2_2 = h2rsqrt(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::sin(h2);
  h2_2 = h2sin(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::sqrt(h2);
  h2_2 = h2sqrt(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::trunc(h2);
  h2_2 = h2trunc(h2);
}

__global__ void kernelFuncInt() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  // CHECK: ll = dpct::max(ll, ll);
  // CHECK-NEXT: ll = dpct::min(ll, ll);
  // CHECK-NEXT: ull = dpct::max(ull, ull);
  // CHECK-NEXT: ull = dpct::min(ull, ull);
  // CHECK-NEXT: u = dpct::max(u, u);
  // CHECK-NEXT: u = dpct::min(u, u);
  ll = llmax(ll, ll);
  ll = llmin(ll, ll);
  ull = ullmax(ull, ull);
  ull = ullmin(ull, ull);
  u = umax(u, u);
  u = umin(u, u);
}

__global__ void kernelFuncSIMD() {
  unsigned int u, u_1, u_2;

  // CHECK: u_2 = sycl::ext::intel::math::vabs2(u);
  u_2 = __vabs2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabs4(u);
  u_2 = __vabs4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffs2(u, u_1);
  u_2 = __vabsdiffs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffs4(u, u_1);
  u_2 = __vabsdiffs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffu2(u, u_1);
  u_2 = __vabsdiffu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsdiffu4(u, u_1);
  u_2 = __vabsdiffu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vabsss2(u);
  u_2 = __vabsss2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vabsss4(u);
  u_2 = __vabsss4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vadd2(u, u_1);
  u_2 = __vadd2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vadd4(u, u_1);
  u_2 = __vadd4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddss2(u, u_1);
  u_2 = __vaddss2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddss4(u, u_1);
  u_2 = __vaddss4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddus2(u, u_1);
  u_2 = __vaddus2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vaddus4(u, u_1);
  u_2 = __vaddus4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgs2(u, u_1);
  u_2 = __vavgs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgs4(u, u_1);
  u_2 = __vavgs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgu2(u, u_1);
  u_2 = __vavgu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vavgu4(u, u_1);
  u_2 = __vavgu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpeq2(u, u_1);
  u_2 = __vcmpeq2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpeq4(u, u_1);
  u_2 = __vcmpeq4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpges2(u, u_1);
  u_2 = __vcmpges2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpges4(u, u_1);
  u_2 = __vcmpges4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgeu2(u, u_1);
  u_2 = __vcmpgeu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgeu4(u, u_1);
  u_2 = __vcmpgeu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgts2(u, u_1);
  u_2 = __vcmpgts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgts4(u, u_1);
  u_2 = __vcmpgts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgtu2(u, u_1);
  u_2 = __vcmpgtu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpgtu4(u, u_1);
  u_2 = __vcmpgtu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmples2(u, u_1);
  u_2 = __vcmples2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmples4(u, u_1);
  u_2 = __vcmples4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpleu2(u, u_1);
  u_2 = __vcmpleu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpleu4(u, u_1);
  u_2 = __vcmpleu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmplts2(u, u_1);
  u_2 = __vcmplts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmplts4(u, u_1);
  u_2 = __vcmplts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpltu2(u, u_1);
  u_2 = __vcmpltu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpltu4(u, u_1);
  u_2 = __vcmpltu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpne2(u, u_1);
  u_2 = __vcmpne2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vcmpne4(u, u_1);
  u_2 = __vcmpne4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vhaddu2(u, u_1);
  u_2 = __vhaddu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vhaddu4(u, u_1);
  u_2 = __vhaddu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxs2(u, u_1);
  u_2 = __vmaxs2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxs4(u, u_1);
  u_2 = __vmaxs4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxu2(u, u_1);
  u_2 = __vmaxu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmaxu4(u, u_1);
  u_2 = __vmaxu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmins2(u, u_1);
  u_2 = __vmins2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vmins4(u, u_1);
  u_2 = __vmins4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vminu2(u, u_1);
  u_2 = __vminu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vminu4(u, u_1);
  u_2 = __vminu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vneg2(u);
  u_2 = __vneg2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vneg4(u);
  u_2 = __vneg4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vnegss2(u);
  u_2 = __vnegss2(u);
  // CHECK: u_2 = sycl::ext::intel::math::vnegss4(u);
  u_2 = __vnegss4(u);
  // CHECK: u_2 = sycl::ext::intel::math::vsads2(u, u_1);
  u_2 = __vsads2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsads4(u, u_1);
  u_2 = __vsads4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsadu2(u, u_1);
  u_2 = __vsadu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsadu4(u, u_1);
  u_2 = __vsadu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vseteq2(u, u_1);
  u_2 = __vseteq2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vseteq4(u, u_1);
  u_2 = __vseteq4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetges2(u, u_1);
  u_2 = __vsetges2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetges4(u, u_1);
  u_2 = __vsetges4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgeu2(u, u_1);
  u_2 = __vsetgeu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgeu4(u, u_1);
  u_2 = __vsetgeu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgts2(u, u_1);
  u_2 = __vsetgts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgts4(u, u_1);
  u_2 = __vsetgts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgtu2(u, u_1);
  u_2 = __vsetgtu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetgtu4(u, u_1);
  u_2 = __vsetgtu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetles2(u, u_1);
  u_2 = __vsetles2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetles4(u, u_1);
  u_2 = __vsetles4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetleu2(u, u_1);
  u_2 = __vsetleu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetleu4(u, u_1);
  u_2 = __vsetleu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetlts2(u, u_1);
  u_2 = __vsetlts2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetlts4(u, u_1);
  u_2 = __vsetlts4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetltu2(u, u_1);
  u_2 = __vsetltu2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetltu4(u, u_1);
  u_2 = __vsetltu4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetne2(u, u_1);
  u_2 = __vsetne2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsetne4(u, u_1);
  u_2 = __vsetne4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsub2(u, u_1);
  u_2 = __vsub2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsub4(u, u_1);
  u_2 = __vsub4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubss2(u, u_1);
  u_2 = __vsubss2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubss4(u, u_1);
  u_2 = __vsubss4(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubus2(u, u_1);
  u_2 = __vsubus2(u, u_1);
  // CHECK: u_2 = sycl::ext::intel::math::vsubus4(u, u_1);
  u_2 = __vsubus4(u, u_1);
}

int main() { return 0; }
