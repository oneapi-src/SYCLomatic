#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/math/cuda-math-intrinsics %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/math/cuda-math-intrinsics/cuda-math-intrinsics.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/math/cuda-math-intrinsics/cuda-math-intrinsics.dp.cpp -o %T/math/cuda-math-intrinsics/cuda-math-intrinsics.dp.o %}

#include <cmath>
#include <iomanip>
#include <iostream>
#include <limits>
#include <algorithm>
#include <complex>

#include <stdio.h>

// CHECK: #include <algorithm>

#include "hip/hip_fp16.h"

using namespace std;

// CHECK: // AAA
// CHECK-EMPTY:
// CHECK-NEXT: // BBB
// AAA
using ::max;
// BBB

// CHECK: static dpct::constant_memory<double, 0> d;
// CHECK-NEXT: static dpct::constant_memory<double, 0> d2;
__constant__ double d;
__constant__ double d2;

// CHECK: double test(double d3, double d) {
// CHECK-NEXT:  return sycl::max(d, d3);
// CHECK-NEXT:}
__device__ double test(double d3) {
  return max(d, d3);
}

// CHECK:  double test2(double d, double d2) {
// CHECK-NEXT:   return sycl::max(d, d2);
// CHECK-NEXT: }
__device__ double test2() {
  return max(d, d2);
}

// CHECK:  double test3(double d4, double d5) {
// CHECK-NEXT:   return sycl::max(d4, d5);
// CHECK-NEXT: }
__device__ double test3(double d4, double d5) {
  return max(d4, d5);
}

// CHECK: static dpct::constant_memory<float, 0> C;
// CHECK-NEXT:  int foo(int n, float C) {
// CHECK-NEXT:   return n == 1 ? C : 0;
// CHECK-NEXT: }
__constant__ float C;
__device__ int foo(int n) {
  return n == 1 ? C : 0;
}

__global__ void kernelFuncHalf(double *deviceArrayDouble) {
  __half h, h_1, h_2;
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = dpct::clamp<sycl::half>(h + h_1, 0.f, 1.0f);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::fma(h, h_1, h_2);
  h_2 = __hfma(h, h_1, h_2);
  // CHECK: h_2 = dpct::clamp<sycl::half>(sycl::fma(h, h_1, h_2), 0.f, 1.0f);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = h * h_1;
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = dpct::clamp<sycl::half>(h * h_1, 0.f, 1.0f);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = -h;
  h_2 = __hneg(h);
  // CHECK: h_2 = h - h_1;
  h_2 = __hsub(h, h_1);
  // CHECK: h_2 = dpct::clamp<sycl::half>(h - h_1, 0.f, 1.0f);
  h_2 = __hsub_sat(h, h_1);

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = h2 + h2_1;
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = dpct::clamp<sycl::half2>(h2 + h2_1, {0.f, 0.f}, {1.f, 1.f});
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::fma(h2, h2_1, h2_2);
  h2_2 = __hfma2(h2, h2_1, h2_2);
  // CHECK: h2_2 = dpct::clamp<sycl::half2>(sycl::fma(h2, h2_1, h2_2), {0.f, 0.f}, {1.f, 1.f});
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = h2 * h2_1;
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = dpct::clamp<sycl::half2>(h2 * h2_1, {0.f, 0.f}, {1.f, 1.f});
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = -h2;
  h2_2 = __hneg2(h2);
  // CHECK: h2_2 = h2 - h2_1;
  h2_2 = __hsub2(h2, h2_1);
  // CHECK: h2_2 = dpct::clamp<sycl::half2>(h2 - h2_1, {0.f, 0.f}, {1.f, 1.f});
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half Comparison Functions

  // CHECK: b = h == h_1;
  b = __heq(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::equal_to<>());
  b = __hequ(h, h_1);
  // CHECK: b = h >= h_1;
  b = __hge(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::greater_equal<>());
  b = __hgeu(h, h_1);
  // CHECK: b = h > h_1;
  b = __hgt(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::greater<>());
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::isinf(h);
  b = __hisinf(h);
  // CHECK: b = sycl::isnan(h);
  b = __hisnan(h);
  // CHECK: b = h <= h_1;
  b = __hle(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::less_equal<>());
  b = __hleu(h, h_1);
  // CHECK: b = h < h_1;
  b = __hlt(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::less<>());
  b = __hltu(h, h_1);
  // CHECK: b = dpct::compare(h, h_1, std::not_equal_to<>());
  b = __hne(h, h_1);
  // CHECK: b = dpct::unordered_compare(h, h_1, std::not_equal_to<>());
  b = __hneu(h, h_1);

  // Half2 Comparison Functions

  // CHECK: b = dpct::compare_both(h2, h2_1, std::equal_to<>());
  b = __hbeq2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::equal_to<>());
  b = __hbequ2(h2, h2_1);
  // CHECK: b = dpct::compare_both(h2, h2_1, std::greater_equal<>());
  b = __hbge2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::greater_equal<>());
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = dpct::compare_both(h2, h2_1, std::greater<>());
  b = __hbgt2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::greater<>());
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = dpct::compare_both(h2, h2_1, std::less_equal<>());
  b = __hble2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::less_equal<>());
  b = __hbleu2(h2, h2_1);
  // CHECK: b = dpct::compare_both(h2, h2_1, std::less<>());
  b = __hblt2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::less<>());
  b = __hbltu2(h2, h2_1);
  // CHECK: b = dpct::compare_both(h2, h2_1, std::not_equal_to<>());
  b = __hbne2(h2, h2_1);
  // CHECK: b = dpct::unordered_compare_both(h2, h2_1, std::not_equal_to<>());
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::equal_to<>());
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::equal_to<>());
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::greater_equal<>());
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::greater_equal<>());
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::greater<>());
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::greater<>());
  h2_2 = __hgtu2(h2, h2_1);
  // CHECK: h2_2 = dpct::isnan(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::less_equal<>());
  h2_2 = __hle2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::less_equal<>());
  h2_2 = __hleu2(h2, h2_1);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::less<>());
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::less<>());
  h2_2 = __hltu2(h2, h2_1);
  // CHECK: h2_2 = dpct::compare(h2, h2_1, std::not_equal_to<>());
  h2_2 = __hne2(h2, h2_1);
  // CHECK: h2_2 = dpct::unordered_compare(h2, h2_1, std::not_equal_to<>());
  h2_2 = __hneu2(h2, h2_1);

  // Half Math Functions

  // CHECK: h_2 = sycl::ceil(h);
  h_2 = hceil(h);
  // CHECK: h_2 = sycl::cos(h);
  h_2 = hcos(h);
  // CHECK: h_2 = sycl::exp(h);
  h_2 = hexp(h);
  // CHECK: h_2 = sycl::exp10(h);
  h_2 = hexp10(h);
  // CHECK: h_2 = sycl::exp2(h);
  h_2 = hexp2(h);
  // CHECK: h_2 = sycl::floor(h);
  h_2 = hfloor(h);
  // CHECK: h_2 = sycl::log(h);
  h_2 = hlog(h);
  // CHECK: h_2 = sycl::log10(h);
  h_2 = hlog10(h);
  // CHECK: h_2 = sycl::log2(h);
  h_2 = hlog2(h);
  // CHECK: h_2 = sycl::half_precision::recip(float(h));
  h_2 = hrcp(h);
  // CHECK: h_2 = sycl::rint(h);
  h_2 = hrint(h);
  // CHECK: h_2 = sycl::rsqrt(h);
  h_2 = hrsqrt(h);
  // CHECK: h_2 = sycl::sin(h);
  h_2 = hsin(h);
  // CHECK: h_2 = sycl::sqrt(h);
  h_2 = hsqrt(h);
  // CHECK: h_2 = sycl::trunc(h);
  h_2 = htrunc(h);

  // Half2 Math Functions

  // CHECK: h2_2 = sycl::ceil(h2);
  h2_2 = h2ceil(h2);
  // CHECK: h2_2 = sycl::cos(h2);
  h2_2 = h2cos(h2);
  // CHECK: h2_2 = sycl::exp(h2);
  h2_2 = h2exp(h2);
  // CHECK: h2_2 = sycl::exp10(h2);
  h2_2 = h2exp10(h2);
  // CHECK: h2_2 = sycl::exp2(h2);
  h2_2 = h2exp2(h2);
  // CHECK: h2_2 = sycl::floor(h2);
  h2_2 = h2floor(h2);
  // CHECK: h2_2 = sycl::log(h2);
  h2_2 = h2log(h2);
  // CHECK: h2_2 = sycl::log10(h2);
  h2_2 = h2log10(h2);
  // CHECK: h2_2 = sycl::log2(h2);
  h2_2 = h2log2(h2);
  // CHECK: h2_2 = sycl::half2(sycl::half_precision::recip(float(h2[0])), sycl::half_precision::recip(float(h2[1])));
  h2_2 = h2rcp(h2);
  // CHECK: h2_2 = sycl::rint(h2);
  h2_2 = h2rint(h2);
  // CHECK: h2_2 = sycl::rsqrt(h2);
  h2_2 = h2rsqrt(h2);
  // CHECK: h2_2 = sycl::sin(h2);
  h2_2 = h2sin(h2);
  // CHECK: h2_2 = sycl::sqrt(h2);
  h2_2 = h2sqrt(h2);
  // CHECK: h2_2 = sycl::trunc(h2);
  h2_2 = h2trunc(h2);
}

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::acos(d0);
  d2 = acos(d0);
  // CHECK: d2 = sycl::acos((double)i);
  d2 = acos(i);

  // CHECK: d2 = sycl::acosh(d0);
  d2 = acosh(d0);
  // CHECK: d2 = sycl::acosh((double)i);
  d2 = acosh(i);

  // CHECK: d2 = sycl::asin(d0);
  d2 = asin(d0);
  // CHECK: d2 = sycl::asin((double)i);
  d2 = asin(i);

  // CHECK: d2 = sycl::asinh(d0);
  d2 = asinh(d0);
  // CHECK: d2 = sycl::asinh((double)i);
  d2 = asinh(i);

  // CHECK: d2 = sycl::atan2(d0, d1);
  d2 = atan2(d0, d1);
  // CHECK: d2 = sycl::atan2((double)i, (double)i);
  d2 = atan2(i, i);
  // CHECK: d2 = sycl::atan2(d0, (double)i);
  d2 = atan2(d0, i);
  // CHECK: d2 = sycl::atan2((double)i, d1);
  d2 = atan2(i, d1);

  // CHECK: d2 = sycl::atan(d0);
  d2 = atan(d0);
  // CHECK: d2 = sycl::atan((double)i);
  d2 = atan(i);

  // CHECK: d2 = sycl::atanh(d0);
  d2 = atanh(d0);
  // CHECK: d2 = sycl::atanh((double)i);
  d2 = atanh(i);

  // CHECK: d2 = sycl::cbrt(d0);
  d2 = cbrt(d0);
  // CHECK: d2 = sycl::cbrt((double)i);
  d2 = cbrt(i);

  // CHECK: d2 = sycl::ceil(d0);
  d2 = ceil(d0);

  // CHECK: d2 = sycl::copysign(d0, d1);
  d2 = copysign(d0, d1);
  // CHECK: d2 = sycl::copysign((double)i, (double)i);
  d2 = copysign(i, i);
  // CHECK: d2 = sycl::copysign(d0, (double)i);
  d2 = copysign(d0, i);
  // CHECK: d2 = sycl::copysign((double)i, d1);
  d2 = copysign(i, d1);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::fabs(d0);
  d2 = fabs(d0);
  // CHECK: d2 = sycl::fabs((double)i);
  d2 = fabs(i);

  // CHECK: sycl::fabs(d0);
  abs(d0);
  // CHECK: sycl::fabs(d0 * d1);
  abs(d0 * d1);

  // CHECK: d2 = sycl::fdim(d0, d1);
  d2 = fdim(d0, d1);
  // CHECK: d2 = sycl::fdim((double)i, (double)i);
  d2 = fdim(i, i);
  // CHECK: d2 = sycl::fdim(d0, (double)i);
  d2 = fdim(d0, i);
  // CHECK: d2 = sycl::fdim((double)i, d1);
  d2 = fdim(i, d1);

  // CHECK: d2 = sycl::floor(d0);
  d2 = floor(d0);
  // CHECK: d2 = sycl::floor((double)i);
  d2 = floor(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::fmax(d0, d1);
  d2 = fmax(d0, d1);
  // CHECK: d2 = sycl::fmax((double)i, (double)i);
  d2 = fmax(i, i);
  // CHECK: d2 = sycl::fmax(d0, (double)i);
  d2 = fmax(d0, i);
  // CHECK: d2 = sycl::fmax((double)i, d1);
  d2 = fmax(i, d1);

  // CHECK: d2 = sycl::fmin(d0, d1);
  d2 = fmin(d0, d1);
  // CHECK: d2 = sycl::fmin((double)i, (double)i);
  d2 = fmin(i, i);
  // CHECK: d2 = sycl::fmin(d0, (double)i);
  d2 = fmin(d0, i);
  // CHECK: d2 = sycl::fmin((double)i, d1);
  d2 = fmin(i, d1);

  // CHECK: d2 = sycl::fmod(d0, d1);
  d2 = fmod(d0, d1);
  // CHECK: d2 = sycl::fmod((double)i, (double)i);
  d2 = fmod(i, i);
  // CHECK: d2 = sycl::fmod(d0, (double)i);
  d2 = fmod(d0, i);
  // CHECK: d2 = sycl::fmod((double)i, d1);
  d2 = fmod(i, d1);

  // CHECK: d2 = sycl::frexp(d0, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  d2 = frexp(d0, &i);
  // CHECK: d2 = sycl::frexp((double)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  d2 = frexp(i, &i);

  // CHECK: d2 = sycl::hypot(d0, d1);
  d2 = hypot(d0, d1);
  // CHECK: d2 = sycl::hypot((double)i, (double)i);
  d2 = hypot(i, i);
  // CHECK: d2 = sycl::hypot(d0, (double)i);
  d2 = hypot(d0, i);
  // CHECK: d2 = sycl::hypot((double)i, d1);
  d2 = hypot(i, d1);

  // CHECK: d2 = sycl::ilogb(d0);
  d2 = ilogb(d0);
  // CHECK: d2 = sycl::ilogb((double)i);
  d2 = ilogb(i);

  // CHECK: d2 = sycl::ldexp(d0, i);
  d2 = ldexp(d0, i);
  // CHECK: d2 = sycl::ldexp((double)i, i);
  d2 = ldexp(i, i);

  // CHECK: d2 = sycl::lgamma(d0);
  d2 = lgamma(d0);
  // CHECK: d2 = sycl::lgamma((double)i);
  d2 = lgamma(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = llrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = llrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = llround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = llround(i);

  // CHECK: d2 = sycl::log10(d0);
  d2 = log10(d0);
  // CHECK: d2 = sycl::log10((double)i);
  d2 = log10(i);

  // CHECK: d2 = sycl::log1p(d0);
  d2 = log1p(d0);
  // CHECK: d2 = sycl::log1p((double)i);
  d2 = log1p(i);

  // CHECK: d2 = sycl::log2(d0);
  d2 = log2(d0);
  // CHECK: d2 = sycl::log2((double)i);
  d2 = log2(i);

  // CHECK: d2 = sycl::logb(d0);
  d2 = logb(d0);
  // CHECK: d2 = sycl::logb((double)i);
  d2 = logb(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = lrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = lrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = lround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = lround(i);

  // CHECK: d2 = sycl::modf(d0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(&d1));
  d2 = modf(d0, &d1);
  // CHECK: d2 = sycl::modf((double)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(&d1));
  d2 = modf(i, &d1);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("");

  // CHECK: d2 = dpct::pow(d0, d1);
  d2 = pow(d0, d1);
  // CHECK: d2 = dpct::pow(i, i);
  d2 = pow(i, i);
  // CHECK: d2 = dpct::pow(d0, i);
  d2 = pow(d0, i);
  // CHECK: d2 = dpct::pow(i, d1);
  d2 = pow(i, d1);

  // CHECK: dpct::pow(f, 1);
  float f;
  pow(f, 1);

  // CHECK: d2 = sycl::remainder(d0, d1);
  d2 = remainder(d0, d1);
  // CHECK: d2 = sycl::remainder((double)i, (double)i);
  d2 = remainder(i, i);
  // CHECK: d2 = sycl::remainder(d0, (double)i);
  d2 = remainder(d0, i);
  // CHECK: d2 = sycl::remainder((double)i, d1);
  d2 = remainder(i, d1);

  // CHECK: d2 = sycl::remquo(d0, d1, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  d2 = remquo(d0, d1, &i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = rint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = rint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = round(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = round(i);

  // CHECK: d2 = sycl::rsqrt(d0);
  d2 = rsqrt(d0);
  // CHECK: d2 = sycl::rsqrt((double)i);
  d2 = rsqrt((double)i);

  // CHECK: d1 = sycl::sincos(d0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(&d2));
  sincos(d0, &d1, &d2);
  // CHECK: d1 = sycl::sincos((double)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(&d2));
  sincos(i, &d1, &d2);

  // CHECK: d2 = sycl::sin(d0);
  d2 = sin(d0);
  // CHECK: d2 = sycl::sin((double)i);
  d2 = sin(i);

  // CHECK: d2 = sycl::sinh(d0);
  d2 = sinh(d0);
  // CHECK: d2 = sycl::sinh((double)i);
  d2 = sinh(i);

  // CHECK: d2 = sycl::sinpi(d0);
  d2 = sinpi(d0);
  // CHECK: d2 = sycl::sinpi((double)i);
  d2 = sinpi((double)i);

  // CHECK: d2 = sycl::sqrt(d0);
  d2 = sqrt(d0);
  // CHECK: d2 = sycl::sqrt((double)i);
  d2 = sqrt(i);

  // CHECK: d2 = sycl::tan(d0);
  d2 = tan(d0);
  // CHECK: d2 = sycl::tan((double)i);
  d2 = tan(i);

  // CHECK: d2 = sycl::tanh(d0);
  d2 = tanh(d0);
  // CHECK: d2 = sycl::tanh((double)i);
  d2 = tanh(i);

  // CHECK: d2 = sycl::tgamma(d0);
  d2 = tgamma(d0);
  // CHECK: d2 = sycl::tgamma((double)i);
  d2 = tgamma(i);

  // CHECK: d2 = sycl::trunc(d0);
  d2 = trunc(d0);
  // CHECK: d2 = sycl::trunc((double)i);
  d2 = trunc(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rn(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rz(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/(d0+d0));
  d1 = __drcp_rz(d0+d0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rn(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rz(d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rd(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rn(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_ru(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rz(d0, d1, d2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rz(i, i, i);

  // CHECK: d0 = sycl::fmin(d0, d1);
  d0 = fmin(d0, d1);
  // CHECK: d0 = sycl::fmin((double)i, (double)i);
  d0 = fmin(i, i);
  // CHECK: d0 = sycl::fmin(d0, (double)i);
  d0 = fmin(d0, i);
  // CHECK: d0 = sycl::fmin((double)i, d1);
  d0 = fmin(i, d1);

  // CHECK: d0 = sycl::fmax(d0, d1);
  d0 = fmax(d0, d1);
  // CHECK: d0 = sycl::fmax((double)i, (double)i);
  d0 = fmax(i, i);
  // CHECK: d0 = sycl::fmax(d0, (double)i);
  d0 = fmax(d0, i);
  // CHECK: d0 = sycl::fmax((double)i, d1);
  d0 = fmax(i, d1);

  // CHECK: d1 = sycl::floor(d1);
  d1 = floor(d1);
  // CHECK: d1 = sycl::floor((double)i);
  d1 = floor(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("NaN");

  // CHECK: d0 = sycl::nextafter(d0, d0);
  d0 = nextafter(d0, d0);
  // CHECK: d0 = sycl::nextafter((double)i, (double)i);
  d0 = nextafter(i, i);
  // CHECK: d0 = sycl::nextafter(d0, (double)i);
  d0 = nextafter(d0, i);
  // CHECK: d0 = sycl::nextafter((double)i, d1);
  d0 = nextafter(i, d1);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::log(f0);
  f2 = logf(f0);
  // CHECK: f2 = sycl::log((float)i);
  f2 = logf(i);

  // CHECK: f2 = sycl::acos(f0);
  f2 = acosf(f0);
  // CHECK: f2 = sycl::acos((float)i);
  f2 = acosf(i);

  // CHECK: f2 = sycl::acosh(f0);
  f2 = acoshf(f0);
  // CHECK: f2 = sycl::acosh((float)i);
  f2 = acoshf(i);

  // CHECK: f2 = sycl::asin(f0);
  f2 = asinf(f0);
  // CHECK: f2 = sycl::asin((float)i);
  f2 = asinf(i);

  // CHECK: f2 = sycl::asinh(f0);
  f2 = asinhf(f0);
  // CHECK: f2 = sycl::asinh((float)i);
  f2 = asinhf(i);

  // CHECK: f2 = sycl::atan2(f0, f1);
  f2 = atan2f(f0, f1);
  // CHECK: f2 = sycl::atan2((float)i, (float)i);
  f2 = atan2f(i, i);
  // CHECK: f2 = sycl::atan2(f0, (float)i);
  f2 = atan2f(f0, i);
  // CHECK: f2 = sycl::atan2((float)i, f1);
  f2 = atan2f(i, f1);

  // CHECK: f2 = sycl::atan(f0);
  f2 = atanf(f0);
  // CHECK: f2 = sycl::atan((float)i);
  f2 = atanf(i);

  // CHECK: f2 = sycl::atanh(f0);
  f2 = atanhf(f0);
  // CHECK: f2 = sycl::atanh((float)i);
  f2 = atanhf(i);

  // CHECK: f2 = sycl::cbrt(f0);
  f2 = cbrtf(f0);
  // CHECK: f2 = sycl::cbrt((float)i);
  f2 = cbrtf(i);

  // CHECK: f2 = sycl::ceil(f0);
  f2 = ceilf(f0);

  // CHECK: f2 = sycl::copysign(f0, f1);
  f2 = copysignf(f0, f1);
  // CHECK: f2 = sycl::copysign((float)i, (float)i);
  f2 = copysignf(i, i);
  // CHECK: f2 = sycl::copysign(f0, (float)i);
  f2 = copysignf(f0, i);
  // CHECK: f2 = sycl::copysign((float)i, f1);
  f2 = copysignf(i, f1);

  // CHECK: f2 = sycl::cos(f0);
  f2 = cosf(f0);
  // CHECK: f2 = sycl::cos((float)i);
  f2 = cosf(i);

  // CHECK: f2 = sycl::cosh(f0);
  f2 = coshf(f0);
  // CHECK: f2 = sycl::cosh((float)i);
  f2 = coshf(i);

  // CHECK: f2 = sycl::cospi(f0);
  f2 = cospif(f0);
  // CHECK: f2 = sycl::cospi((float)i);
  f2 = cospif(i);

  // CHECK: f2 = sycl::erfc(f0);
  f2 = erfcf(f0);
  // CHECK: f2 = sycl::erfc((float)i);
  f2 = erfcf(i);

  // CHECK: f2 = sycl::erf(f0);
  f2 = erff(f0);
  // CHECK: f2 = sycl::erf((float)i);
  f2 = erff(i);

  // CHECK: f2 = sycl::exp10(f0);
  f2 = exp10f(f0);
  // CHECK: f2 = sycl::exp10((float)i);
  f2 = exp10f(i);

  // CHECK: f2 = sycl::exp2(f0);
  f2 = exp2f(f0);
  // CHECK: f2 = sycl::exp2((float)i);
  f2 = exp2f(i);

  // CHECK: f2 = sycl::native::exp(f0);
  f2 = expf(f0);
  // CHECK: f2 = sycl::native::exp((float)i);
  f2 = expf(i);

  // CHECK: f2 = sycl::expm1(f0);
  f2 = expm1f(f0);
  // CHECK: f2 = sycl::expm1((float)i);
  f2 = expm1f(i);

  // CHECK: f2 = sycl::fabs(f0);
  f2 = fabsf(f0);
  // CHECK: f2 = sycl::fabs((float)i);
  f2 = fabsf(i);

  // CHECK: f2 = sycl::fdim(f0, f1);
  f2 = fdimf(f0, f1);
  // CHECK: f2 = sycl::fdim((float)i, (float)i);
  f2 = fdimf(i, i);
  // CHECK: f2 = sycl::fdim(f0, (float)i);
  f2 = fdimf(f0, i);
  // CHECK: f2 = sycl::fdim((float)i, f1);
  f2 = fdimf(i, f1);

  // CHECK: f2 = f0 / f1;
  f2 = fdividef(f0, f1);
  // CHECK: f2 = i / i;
  f2 = fdividef(i, i);
  // CHECK: f2 = f0 / i;
  f2 = fdividef(f0, i);
  // CHECK: f2 = i / f1;
  f2 = fdividef(i, f1);

  // CHECK: f2 = sycl::floor(f0);
  f2 = floorf(f0);
  // CHECK: f2 = sycl::floor((float)i);
  f2 = floorf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f2 = sycl::fmin(f0, f1);
  f2 = fminf(f0, f1);
  // CHECK: f2 = sycl::fmin((float)i, (float)i);
  f2 = fminf(i, i);
  // CHECK: f2 = sycl::fmin(f0, (float)i);
  f2 = fminf(f0, i);
  // CHECK: f2 = sycl::fmin((float)i, f1);
  f2 = fminf(i, f1);

  // CHECK: f2 = sycl::fmod(f0, f1);
  f2 = fmodf(f0, f1);
  // CHECK: f2 = sycl::fmod((float)i, (float)i);
  f2 = fmodf(i, i);
  // CHECK: f2 = sycl::fmod(f0, (float)i);
  f2 = fmodf(f0, i);
  // CHECK: f2 = sycl::fmod((float)i, f1);
  f2 = fmodf(i, f1);

  // CHECK: f2 = sycl::frexp(f0, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = frexpf(f0, &i);
  // CHECK: f2 = sycl::frexp((float)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = frexpf(i, &i);

  // CHECK: f2 = sycl::hypot(f0, f1);
  f2 = hypotf(f0, f1);
  // CHECK: f2 = sycl::hypot((float)i, (float)i);
  f2 = hypotf(i, i);
  // CHECK: f2 = sycl::hypot(f0, (float)i);
  f2 = hypotf(f0, i);
  // CHECK: f2 = sycl::hypot((float)i, f1);
  f2 = hypotf(i, f1);

  // CHECK: f2 = sycl::ilogb(f0);
  f2 = ilogbf(f0);
  // CHECK: f2 = sycl::ilogb((float)i);
  f2 = ilogbf(i);

  // CHECK: i = sycl::isfinite(f0);
  i = isfinite(f0);
  // CHECK: i = sycl::isfinite((float)i);
  i = isfinite(i);

  // CHECK: i = sycl::isinf(f0);
  i = isinf(f0);
  // CHECK: i = sycl::isinf((float)i);
  i = isinf(i);

  // CHECK: i = sycl::isnan(f0);
  i = isnan(f0);
  // CHECK: i = sycl::isnan((float)i);
  i = isnan(i);

  // CHECK: f2 = sycl::ldexp(f0, i);
  f2 = ldexpf(f0, i);
  // CHECK: f2 = sycl::ldexp((float)i, i);
  f2 = ldexpf(i, i);

  // CHECK: f2 = sycl::lgamma(f0);
  f2 = lgammaf(f0);
  // CHECK: f2 = sycl::lgamma((float)i);
  f2 = lgammaf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = llrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = llrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = llroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = llroundf(i);

  // CHECK: f2 = sycl::log10(f0);
  f2 = log10f(f0);
  // CHECK: f2 = sycl::log10((float)i);
  f2 = log10f(i);

  // CHECK: f2 = sycl::log1p(f0);
  f2 = log1pf(f0);
  // CHECK: f2 = sycl::log1p((float)i);
  f2 = log1pf(i);

  // CHECK: f2 = sycl::log2(f0);
  f2 = log2f(f0);
  // CHECK: f2 = sycl::log2((float)i);
  f2 = log2f(i);

  // CHECK: f2 = sycl::logb(f0);
  f2 = logbf(f0);
  // CHECK: f2 = sycl::logb((float)i);
  f2 = logbf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = lrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = lrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = lroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = lroundf(i);

  // CHECK: f2 = sycl::modf(f0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f1));
  f2 = modff(f0, &f1);
  // CHECK: f2 = sycl::modf((float)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f1));
  f2 = modff(i, &f1);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nan("");

  // CHECK: f2 = dpct::pow(f0, f1);
  f2 = powf(f0, f1);
  // CHECK: f2 = dpct::pow(i, i);
  f2 = powf(i, i);
  // CHECK: f2 = dpct::pow(f0, i);
  f2 = powf(f0, i);
  // CHECK: f2 = dpct::pow(i, f1);
  f2 = powf(i, f1);

  // CHECK: f2 = sycl::remainder(f0, f1);
  f2 = remainderf(f0, f1);
  // CHECK: f2 = sycl::remainder((float)i, (float)i);
  f2 = remainderf(i, i);
  // CHECK: f2 = sycl::remainder(f0, (float)i);
  f2 = remainderf(f0, i);
  // CHECK: f2 = sycl::remainder((float)i, f1);
  f2 = remainderf(i, f1);

  // CHECK: f2 = sycl::remquo(f0, f1, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = remquof(f0, f1, &i);
  // CHECK: f2 = sycl::remquo((float)i, (float)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = remquof(i, i, &i);
  // CHECK: f2 = sycl::remquo(f0, (float)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = remquof(f0, i, &i);
  // CHECK: f2 = sycl::remquo((float)i, f1, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  f2 = remquof(i, f1, &i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = rintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = rintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = roundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = roundf(i);

  // CHECK: f2 = sycl::rsqrt(f0);
  f2 = rsqrtf(f0);
  // CHECK: f2 = sycl::rsqrt((float)i);
  f2 = rsqrtf(i);

  // CHECK: f2 = sycl::signbit(f0);
  f2 = signbit(f0);
  // CHECK: f2 = sycl::signbit((float)i);
  f2 = signbit(i);

  // CHECK: f1 = sycl::sincos(f0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f2));
  sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f2));
  sincosf(i, &f1, &f2);

  // CHECK: f2 = sycl::sin(f0);
  f2 = sinf(f0);
  // CHECK: f2 = sycl::sin((float)i);
  f2 = sinf(i);

  // CHECK: f2 = sycl::sinh(f0);
  f2 = sinhf(f0);
  // CHECK: f2 = sycl::sinh((float)i);
  f2 = sinhf(i);

  // CHECK: f2 = sycl::sinpi(f0);
  f2 = sinpif(f0);
  // CHECK: f2 = sycl::sinpi((float)i);
  f2 = sinpif(i);

  // CHECK: f2 = sycl::sqrt(f0);
  f2 = sqrtf(f0);
  // CHECK: f2 = sycl::sqrt((float)i);
  f2 = sqrtf(i);

  // CHECK: f2 = sycl::tan(f0);
  f2 = tanf(f0);
  // CHECK: f2 = sycl::tan((float)i);
  f2 = tanf(i);

  // CHECK: f2 = sycl::tanh(f0);
  f2 = tanhf(f0);
  // CHECK: f2 = sycl::tanh((float)i);
  f2 = tanhf(i);

  // CHECK: f2 = sycl::tgamma(f0);
  f2 = tgammaf(f0);
  // CHECK: f2 = sycl::tgamma((float)i);
  f2 = tgammaf(i);

  // CHECK: f2 = sycl::trunc(f0);
  f2 = truncf(f0);
  // CHECK: f2 = sycl::trunc((float)i);
  f2 = truncf(i);

  // CHECK: f0 = sycl::cos(f0);
  f0 = __cosf(f0);
  // CHECK: f0 = sycl::cos((float)i);
  f0 = __cosf(i);

  // CHECK: f0 = sycl::exp10(f0);
  f0 = __exp10f(f0);
  // CHECK: f0 = sycl::exp10((float)i);
  f0 = __exp10f(i);

  // CHECK: f0 = sycl::native::exp(f0);
  f0 = __expf(f0);
  // CHECK: f0 = sycl::native::exp((float)i);
  f0 = __expf(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rz(f0, f1);

  // CHECK: f2 = f0 / f1;
  f2 = __fdividef(f0, f1);
  // CHECK: f2 = i / i;
  f2 = __fdividef(i, i);
  // CHECK: f2 = f0 / i;
  f2 = __fdividef(f0, i);
  // CHECK: f2 = i / f1;
  f2 = __fdividef(i, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rd(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rn(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_ru(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rz(f0, f1, f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rz(i, i, i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rn(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rz(f0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rn(f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rz(f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rz(f0, f1);

  // CHECK: f1 = sycl::log10(f1);
  f1 = __log10f(f1);
  // CHECK: f1 = sycl::log10((float)i);
  f1 = __log10f(i);

  // CHECK: f1 = sycl::log2(f1);
  f1 = __log2f(f1);
  // CHECK: f1 = sycl::log2((float)i);
  f1 = __log2f(i);

  // CHECK: f1 = sycl::log(f1);
  f1 = __logf(f1);
  // CHECK: f1 = sycl::log((float)i);
  f1 = __logf(i);

  // CHECK: f2 = dpct::pow(f0, f1);
  f2 = __powf(f0, f1);
  // CHECK: f2 = dpct::pow(i, i);
  f2 = __powf(i, i);
  // CHECK: f2 = dpct::pow(f0, i);
  f2 = __powf(f0, i);
  // CHECK: f2 = dpct::pow(i, f1);
  f2 = __powf(i, f1);

  // CHECK: f1 = sycl::sincos(f0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f2));
  __sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, float>(&f2));
  __sincosf(i, &f1, &f2);

  // CHECK: f1 = sycl::sin(f1);
  f1 = __sinf(f1);
  // CHECK: f1 = sycl::sin((float)i);
  f1 = __sinf(i);

  // CHECK: f1 = sycl::tan(f1);
  f1 = __tanf(f1);
  // CHECK: f1 = sycl::tan((float)i);
  f1 = __tanf(i);

  // CHECK: f0 = sycl::fmin(f0, f1);
  f0 = fminf(f0, f1);
  // CHECK: f0 = sycl::fmin((float)i, (float)i);
  f0 = fminf(i, i);
  // CHECK: f0 = sycl::fmin(f0, (float)i);
  f0 = fminf(f0, i);
  // CHECK: f0 = sycl::fmin((float)i, f1);
  f0 = fminf(i, f1);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f1 = sycl::floor(f1);
  f1 = floorf(f1);
  // CHECK: f1 = sycl::floor((float)i);
  f1 = floorf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nanf("NaN");

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt(f2);
  f2 = __frsqrt_rn(f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt((float)i);
  f2 = __frsqrt_rn(i);

  // CHECK: f0 = sycl::nextafter(f0, f0);
  f0 = nextafterf(f0, f0);
  // CHECK: f0 = sycl::nextafter((float)i, (float)i);
  f0 = nextafterf(i, i);
  // CHECK: f0 = sycl::nextafter(f0, (float)i);
  f0 = nextafterf(f0, i);
  // CHECK: f0 = sycl::nextafter((float)i, f1);
  f0 = nextafterf(i, f1);
}

__global__ void kernelFuncTypecasts() {
  short s, s_1;
  unsigned short us;
  int i, i_1;
  unsigned int ui, ui_1;
  long l;
  unsigned long ul;
  long long ll;
  unsigned long long ull;

  __half h;
  __half2 h2;
  float f;
  float2 f2;
  double d;
  double2 d2;

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __double2float_rd(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __double2float_rn(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __double2float_ru(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __double2float_rz(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtn>()[0];
  i = __double2int_rd(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rte>()[0];
  i = __double2int_rn(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtp>()[0];
  i = __double2int_ru(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtz>()[0];
  i = __double2int_rz(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __double2ll_rd(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __double2ll_rn(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __double2ll_ru(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __double2ll_rz(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtn>()[0];
  ui = __double2uint_rd(d);

  // CHECK:ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rte>()[0];
  ui = __double2uint_rn(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtp>()[0];
  ui = __double2uint_ru(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtz>()[0];
  ui = __double2uint_rz(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __double2ull_rd(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __double2ull_rn(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __double2ull_ru(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __double2ull_rz(d);

  // CHECK: ll = sycl::bit_cast<long long>(d);
  ll = __double_as_longlong(d);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtn>()[0];
  i = __float2int_rd(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rte>()[0];
  i = __float2int_rn(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtp>()[0];
  i = __float2int_ru(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtz>()[0];
  i = __float2int_rz(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __float2ll_rd(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __float2ll_rn(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __float2ll_ru(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __float2ll_rz(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtn>()[0];
  ui = __float2uint_rd(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rte>()[0];
  ui = __float2uint_rn(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtp>()[0];
  ui = __float2uint_ru(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtz>()[0];
  ui = __float2uint_rz(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __float2ull_rd(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __float2ull_rn(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __float2ull_ru(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __float2ull_rz(f);

  // CHECK: i = sycl::bit_cast<int>(f);
  i = __float_as_int(f);

  // CHECK: ui = sycl::bit_cast<unsigned int>(f);
  ui = __float_as_uint(f);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __int2double_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtn>()[0];
  d = __int2float_rd(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rte>()[0];
  d = __int2float_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtp>()[0];
  d = __int2float_ru(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtz>()[0];
  d = __int2float_rz(i);

  // CHECK: f = sycl::bit_cast<float>(i);
  f = __int_as_float(i);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtn>()[0];
  d = __ll2double_rd(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __ll2double_rn(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtp>()[0];
  d = __ll2double_ru(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtz>()[0];
  d = __ll2double_rz(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __ll2float_rd(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __ll2float_rn(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __ll2float_ru(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __ll2float_rz(ll);

  // CHECK: d = sycl::bit_cast<double>(ll);
  d = __longlong_as_double(ll);

  // CHECK: d = sycl::vec<unsigned int, 1>{ui}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __uint2double_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __uint2float_rd(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __uint2float_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __uint2float_ru(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __uint2float_rz(ui);

  // CHECK: f = sycl::bit_cast<float>(ui);
  f = __uint_as_float(ui);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtn>()[0];
  d = __ull2double_rd(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __ull2double_rn(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtp>()[0];
  d = __ull2double_ru(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtz>()[0];
  d = __ull2double_rz(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __ull2float_rd(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __ull2float_rn(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __ull2float_ru(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __ull2float_rz(ull);
}

void testDouble() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(double);

  double *hostArrayDouble = (double *)malloc(bytes);
  memset(hostArrayDouble, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayDouble = pi;
  *(hostArrayDouble + 1) = pi - 1;

  double *deviceArrayDouble;
  hipMalloc((double **)&deviceArrayDouble, bytes);

  hipMemcpy(deviceArrayDouble, hostArrayDouble, bytes, hipMemcpyHostToDevice);

  kernelFuncDouble<<<1, 1>>>(deviceArrayDouble);

  hipMemcpy(hostArrayDouble, deviceArrayDouble, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayDouble);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayDouble + 2) << endl;
}

void testFloat() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(float);

  float *hostArrayFloat = (float *)malloc(bytes);
  memset(hostArrayFloat, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayFloat = pi;
  *(hostArrayFloat + 1) = pi - 1;

  float *deviceArrayFloat;
  hipMalloc((float **)&deviceArrayFloat, bytes);

  hipMemcpy(deviceArrayFloat, hostArrayFloat, bytes, hipMemcpyHostToDevice);

  kernelFuncFloat<<<1, 1>>>(deviceArrayFloat);

  hipMemcpy(hostArrayFloat, deviceArrayFloat, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayFloat);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayFloat + 2) << endl;
}

__global__ void testUnsupported() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  half h;
  float f;
  double d;
  half2 h2;
  bool b;

#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i0f is not supported.
  // CHECK-NEXT: */
  f = cyl_bessel_i0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i1f is not supported.
  // CHECK-NEXT: */
  f = cyl_bessel_i1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfcinvf is not supported.
  // CHECK-NEXT: */
  f = erfcinvf(f);
#endif
  // CHECK: f = sycl::exp(f*f)*sycl::erfc(f);
  f = erfcxf(f);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfinvf is not supported.
  // CHECK-NEXT: */
  f = erfinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j0f is not supported.
  // CHECK-NEXT: */
  f = j0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j1f is not supported.
  // CHECK-NEXT: */
  f = j1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of jnf is not supported.
  // CHECK-NEXT: */
  f = jnf(i, f);
#endif

  // CHECK: f = sycl::length(sycl::float3(f, f, f));
  f = norm3df(f, f, f);
  // CHECK: f = sycl::length(sycl::float4(f, f, f, f));
  f = norm4df(f, f, f, f);
  // CHECK: f = sycl::erfc(f / -sycl::sqrt(2.0)) / 2;
  f = normcdff(f);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdfinvf is not supported.
  // CHECK-NEXT: */
  f = normcdfinvf(f);
#endif
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the normf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = dpct::length(&f, i);
  f = normf(i, &f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::cbrt call is used instead of the rcbrtf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::native::recip(dpct::cbrt<float>(f));
  f = rcbrtf(f);
  // CHECK: f = sycl::native::recip(sycl::length(sycl::float3(f, f, f)));
  f = rnorm3df(f, f, f);
  // CHECK: f = sycl::native::recip(sycl::length(sycl::float4(f, f, f, f)));
  f = rnorm4df(f, f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the rnormf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::native::recip(dpct::length(&f, i));
  f = rnormf(i, &f);
  // CHECK: f = f*(2<<l);
  f = scalblnf(f, l);
  // CHECK: f = f*(2<<i);
  f = scalbnf(f, i);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y0f is not supported.
  // CHECK-NEXT: */
  f = y0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y1f is not supported.
  // CHECK-NEXT: */
  f = y1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of ynf is not supported.
  // CHECK-NEXT: */
  f = ynf(i, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i0 is not supported.
  // CHECK-NEXT: */
  d = cyl_bessel_i0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i1 is not supported.
  // CHECK-NEXT: */
  d = cyl_bessel_i1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfcinv is not supported.
  // CHECK-NEXT: */
  d = erfcinv(d);
#endif
  // CHECK: d = sycl::exp(d*d)*sycl::erfc(d);
  d = erfcx(d);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfinv is not supported.
  // CHECK-NEXT: */
  d = erfinv(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j0 is not supported.
  // CHECK-NEXT: */
  d = j0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j1 is not supported.
  // CHECK-NEXT: */
  d = j1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of jn is not supported.
  // CHECK-NEXT: */
  d = jn(i, d);
#endif
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = dpct::length(&d, i);
  d = norm(i, &d);
  // CHECK: d = sycl::length(sycl::double3(d, d, d));
  d = norm3d(d, d, d);
  // CHECK: d = sycl::length(sycl::double4(d, d, d, d));
  d = norm4d(d, d, d, d);
  // CHECK:  d = sycl::erfc(d / -sycl::sqrt(2.0)) / 2;
  d = normcdf(d);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdfinv is not supported.
  // CHECK-NEXT: */
  d = normcdfinv(d);
#endif
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::cbrt call is used instead of the rcbrt call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = 1 / dpct::cbrt<double>(d);
  d = rcbrt(d);
  // CHECK: d = 1 / sycl::length(sycl::double3(d, d, d));
  d = rnorm3d(d, d, d);
  // CHECK: d = 1 / sycl::length(sycl::double4(d, d, d, d));
  d = rnorm4d(d, d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the rnorm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = 1 / dpct::length(&d, i);
  d = rnorm(i, &d);
  // CHECK: d = d*(2<<l);
  d = scalbln(d, l);
  // CHECK: d = d*(2<<i);
  d = scalbn(d, i);
#ifndef BUILD_TEST
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y0 is not supported.
  // CHECK-NEXT: */
  d = y0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y1 is not supported.
  // CHECK-NEXT: */
  d = y1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of yn is not supported.
  // CHECK-NEXT: */
  d = yn(i, d);
#endif

  // CHECK: f = sycl::clamp<float>(f, 0.0f, 1.0f);
  f = __saturatef(f);

  // i = __shfl_down_sync(u, h, u, i);
  // i = __shfl_sync(u, h, u, i);
  // i = __shfl_up_sync(u, h, u, i);
  // i = __shfl_xor_sync(u, h, u, i);

  // CHECK: i = dpct::cast_double_to_int(d);
  i = __double2hiint(d);
  // CHECK: i = dpct::cast_double_to_int(d, false);
  i = __double2loint(d);
  // CHECK: d = dpct::cast_ints_to_double(i, i);
  d = __hiloint2double(i, i);

  // CHECK: u = dpct::reverse_bits<unsigned int>(u);
  u = __brev(u);
  // CHECK: ull = dpct::reverse_bits<unsigned long long>(ull);
  ull = __brevll(ull);
  // CHECK: u = dpct::byte_level_permute(u, u, u);
  u = __byte_perm(u, u, u);
  // CHECK: i = dpct::ffs<int>(i);
  i = __ffs(i);
  // CHECK: i = dpct::ffs<long long int>(ll);
  i = __ffsll(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::funnelshift_l call is used instead of the __funnelshift_l call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = dpct::funnelshift_l(u, u, u);
  u = __funnelshift_l(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::funnelshift_lc call is used instead of the __funnelshift_lc call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = dpct::funnelshift_lc(u, u, u);
  u = __funnelshift_lc(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::funnelshift_r call is used instead of the __funnelshift_r call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = dpct::funnelshift_r(u, u, u);
  u = __funnelshift_r(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::funnelshift_rc call is used instead of the __funnelshift_rc call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = dpct::funnelshift_rc(u, u, u);
  u = __funnelshift_rc(u, u, u);
  // CHECK: ll = sycl::mul_hi(ll, ll);
  ll = __mul64hi(ll, ll);
  // CHECK: i = sycl::rhadd(i, i);
  i = __rhadd(i, i);
  // CHECK: u = sycl::abs_diff(i, i)+u;
  u = __sad(i, i, u);
  // CHECK: u = sycl::hadd(u, u);
  u = __uhadd(u, u);
  // CHECK: u = sycl::mul24(u, u);
  u = __umul24(u, u);
  // CHECK: ull = sycl::mul_hi(ull, ull);
  ull = __umul64hi(ull, ull);
  // CHECK: u = sycl::mul_hi(u, u);
  u = __umulhi(u, u);
  // CHECK: u = sycl::rhadd(u, u);
  u = __urhadd(u, u);
  // CHECK: u = sycl::abs_diff(u, u)+u;
  u = __usad(u, u, u);
  // CHECK: h = h + h;
  h = __hadd(h, h);
}

__global__ void testSimulation() {
  float f;
  double d;

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::floor(f + 0.5);
  f = nearbyintf(f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::floor(d + 0.5);
  d = nearbyint(d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::hypot call is used instead of the rhypotf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = 1 / sycl::hypot(f, f);
  f = rhypotf(f, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospif call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::sincos(f * DPCT_PI_F, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, float>(&f));
  sincospif(f, &f, &f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospi call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::sincos(d * DPCT_PI, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, double>(&d));
  sincospi(d, &d, &d);
}

__global__ void testIntegerFunctions() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;

  // CHECK: i = sycl::clz(i);
  // CHECK-NEXT: i = sycl::clz(ll);
  // CHECK-NEXT: i = sycl::hadd(i, i);
  // CHECK-NEXT: i = sycl::mul24(i, i);
  // CHECK-NEXT: i = sycl::mul_hi(i, i);
  // CHECK-NEXT: i = sycl::popcount(u);
  // CHECK-NEXT: i = sycl::popcount(ull);
  // CHECK-NEXT: i = dpct::dp2a_lo(i, i, 3);
  // CHECK-NEXT: i = dpct::dp2a_hi(i, i, 3);
  // CHECK-NEXT: i = dpct::dp4a(i, i, 3);
  i = __clz(i);
  i = __clzll(ll);
  i = __hadd(i, i);
  i = __mul24(i, i);
  i = __mulhi(i, i);
  i = __popc(u);
  i = __popcll(ull);
  i = __dp2a_lo(i, i, 3);
  i = __dp2a_hi(i, i, 3);
  i = __dp4a(i, i, 3);

  // CHECK: sycl::clz((int)u);
  // CHECK-NEXT: sycl::clz((long long)ull);
  // CHECK-NEXT: sycl::hadd((int)u, (int)u);
  // CHECK-NEXT: sycl::mul24((int)u, (int)u);
  // CHECK-NEXT: sycl::mul_hi((int)u, (int)u);
  __clz(u);
  __clzll(ull);
  __hadd(u, u);
  __mul24(u, u);
  __mulhi(u, u);

  // CHECK: i = sycl::abs(i);
  // CHECK-NEXT: l = sycl::abs(l);
  // CHECK-NEXT: ll = sycl::abs(ll);
  i = abs(i);
  l = labs(l);
  ll = llabs(ll);

  // CHECK: ll = dpct::max(ll, ll);
  // CHECK-NEXT: ll = dpct::min(ll, (long long)l);
  // CHECK-NEXT: ull = dpct::max((unsigned long long)ll, ull);
  // CHECK-NEXT: ull = dpct::min((unsigned long long)ll, (unsigned long long)ll);
  // CHECK-NEXT: u = dpct::max(u, u);
  // CHECK-NEXT: u = dpct::min(u, u);
  ll = llmax(ll, ll);
  ll = llmin(ll, l);
  ull = ullmax(ll, ull);
  ull = ullmin(ll, ll);
  u = umax(u, u);
  u = umin(u, u);
}

__global__ void kernelFuncSIMD() {
  unsigned int u, u_1, u_2;
  bool b;

  // CHECK: u_2 = dpct::vectorized_unary<sycl::short2>(u, dpct::abs());
  // CHECK-NEXT: u_2 = dpct::vectorized_unary<sycl::char4>(u, dpct::abs());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::abs_diff());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::abs_diff());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::abs_diff());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::abs_diff());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::short2>(u, 0, dpct::abs_diff());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, 0, dpct::abs_diff());
  u_2 = __vabs2(u);
  u_2 = __vabs4(u);
  u_2 = __vabsdiffs2(u, u_1);
  u_2 = __vabsdiffs4(u, u_1);
  u_2 = __vabsdiffu2(u, u_1);
  u_2 = __vabsdiffu4(u, u_1);
  u_2 = __vabsss2(u);
  u_2 = __vabsss4(u);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::plus<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::plus<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::add_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::add_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::add_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::add_sat());
  u_2 = __vadd2(u, u_1);
  u_2 = __vadd4(u, u_1);
  u_2 = __vaddss2(u, u_1);
  u_2 = __vaddss4(u, u_1);
  u_2 = __vaddus2(u, u_1);
  u_2 = __vaddus4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::rhadd());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::rhadd());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::rhadd());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::rhadd());
  u_2 = __vavgs2(u, u_1);
  u_2 = __vavgs4(u, u_1);
  u_2 = __vavgu2(u, u_1);
  u_2 = __vavgu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::equal_to<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::equal_to<>());
  u_2 = __vcmpeq2(u, u_1);
  u_2 = __vcmpeq4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::greater_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::greater_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::greater_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::greater_equal<>());
  u_2 = __vcmpges2(u, u_1);
  u_2 = __vcmpges4(u, u_1);
  u_2 = __vcmpgeu2(u, u_1);
  u_2 = __vcmpgeu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::greater<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::greater<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::greater<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::greater<>());
  u_2 = __vcmpgts2(u, u_1);
  u_2 = __vcmpgts4(u, u_1);
  u_2 = __vcmpgtu2(u, u_1);
  u_2 = __vcmpgtu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::less_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::less_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::less_equal<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::less_equal<>());
  u_2 = __vcmples2(u, u_1);
  u_2 = __vcmples4(u, u_1);
  u_2 = __vcmpleu2(u, u_1);
  u_2 = __vcmpleu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::less<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::less<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::less<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::less<>());
  u_2 = __vcmplts2(u, u_1);
  u_2 = __vcmplts4(u, u_1);
  u_2 = __vcmpltu2(u, u_1);
  u_2 = __vcmpltu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::not_equal_to<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::not_equal_to<>());
  u_2 = __vcmpne2(u, u_1);
  u_2 = __vcmpne4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::hadd());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::hadd());
  u_2 = __vhaddu2(u, u_1);
  u_2 = __vhaddu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::maximum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::maximum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::maximum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::maximum());
  u_2 = __vmaxs2(u, u_1);
  u_2 = __vmaxs4(u, u_1);
  u_2 = __vmaxu2(u, u_1);
  u_2 = __vmaxu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::minimum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::minimum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::minimum());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::minimum());
  u_2 = __vmins2(u, u_1);
  u_2 = __vmins4(u, u_1);
  u_2 = __vminu2(u, u_1);
  u_2 = __vminu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_unary<sycl::short2>(u, std::negate<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_unary<sycl::char4>(u, std::negate<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::short2>(0, u, dpct::sub_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(0, u, dpct::sub_sat());
  u_2 = __vneg2(u);
  u_2 = __vneg4(u);
  u_2 = __vnegss2(u);
  u_2 = __vnegss4(u);

  // CHECK: u_2 = dpct::vectorized_sum_abs_diff<sycl::short2>(u, u_1);
  // CHECK-NEXT: u_2 = dpct::vectorized_sum_abs_diff<sycl::char4>(u, u_1);
  // CHECK-NEXT: u_2 = dpct::vectorized_sum_abs_diff<sycl::ushort2>(u, u_1);
  // CHECK-NEXT: u_2 = dpct::vectorized_sum_abs_diff<sycl::uchar4>(u, u_1);
  u_2 = __vsads2(u, u_1);
  u_2 = __vsads4(u, u_1);
  u_2 = __vsadu2(u, u_1);
  u_2 = __vsadu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::equal_to<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::equal_to<unsigned char>());
  u_2 = __vseteq2(u, u_1);
  u_2 = __vseteq4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::greater_equal<short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::greater_equal<char>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::greater_equal<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::greater_equal<unsigned char>());
  u_2 = __vsetges2(u, u_1);
  u_2 = __vsetges4(u, u_1);
  u_2 = __vsetgeu2(u, u_1);
  u_2 = __vsetgeu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::greater<short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::greater<char>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::greater<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::greater<unsigned char>());
  u_2 = __vsetgts2(u, u_1);
  u_2 = __vsetgts4(u, u_1);
  u_2 = __vsetgtu2(u, u_1);
  u_2 = __vsetgtu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::less_equal<short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::less_equal<char>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::less_equal<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::less_equal<unsigned char>());
  u_2 = __vsetles2(u, u_1);
  u_2 = __vsetles4(u, u_1);
  u_2 = __vsetleu2(u, u_1);
  u_2 = __vsetleu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, std::less<short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, std::less<char>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::less<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::less<unsigned char>());
  u_2 = __vsetlts2(u, u_1);
  u_2 = __vsetlts4(u, u_1);
  u_2 = __vsetltu2(u, u_1);
  u_2 = __vsetltu4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::not_equal_to<unsigned short>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::not_equal_to<unsigned char>());
  u_2 = __vsetne2(u, u_1);
  u_2 = __vsetne4(u, u_1);

  // CHECK: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, std::minus<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, std::minus<>());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::short2>(u, u_1, dpct::sub_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::char4>(u, u_1, dpct::sub_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::ushort2>(u, u_1, dpct::sub_sat());
  // CHECK-NEXT: u_2 = dpct::vectorized_binary<sycl::uchar4>(u, u_1, dpct::sub_sat());
  u_2 = __vsub2(u, u_1);
  u_2 = __vsub4(u, u_1);
  u_2 = __vsubss2(u, u_1);
  u_2 = __vsubss4(u, u_1);
  u_2 = __vsubus2(u, u_1);
  u_2 = __vsubus4(u, u_1);
}

void testTypecasts() {

}

__global__ void testConditionalOperator(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1),
        &f2 = *(deviceArrayFloat + 2);
  // CHECK: f0 = sycl::fmax(f0 = (f1) > (f1 == 1 ? 0 : -f2) ? f1 * f1 / f1 : -f1, f1 + f1 < f2
  // CHECK-NEXT:         ? ((f1) > (f1 == 1 ? 0 : -f2) ? f2 * f2 / f1 : -f1)
  // CHECK-NEXT:         : -f1);
  // CHECK-NEXT: f0 = f1 > f2 ? f1 * f1 / f1 : f1;
  // CHECK-NEXT: f0 = sycl::fmax(0 ? f1 * f1 / f1 : f1, f2);
  f0 = fmaxf(
      f0 = (f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f1, 2.f), f1) : -f1,
      f1 + f1 < f2
          ? ((f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f2, 2.f), f1) : -f1)
          : -f1);
  f0 = f1 > f2 ? __fdividef(__powf(f1, 2.f), f1) : f1;
  f0 = fmaxf(0 ? __fdividef(__powf(f1, 2.f), f1) : f1, f2);
}

int main() {
  testDouble();
  testFloat();
  testTypecasts();
}

// CHECK:  int foo(int i, int j) {
// CHECK-NEXT:   return std::max(i, j) + std::min(i, j);
// CHECK-NEXT: }
__host__ int foo(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo(float f, float g) {
// CHECK-NEXT:   return max(f, g) + min(f, g);
// CHECK-NEXT: }
__host__ float foo(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int foo2(int i, int j) {
// CHECK-NEXT:   return sycl::max(i, j) + sycl::min(i, j);
// CHECK-NEXT: }
__device__ int foo2(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo2(float f, float g) {
// CHECK-NEXT:   return sycl::max(f, g) + sycl::min(f, g);
// CHECK-NEXT: }
__device__ float foo2(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int  foo3(int i, int j) {
// CHECK-NEXT:   return std::max(i, j) + std::min(i, j);
// CHECK-NEXT: }
__device__ int __host__ foo3(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float  foo3(float f, float g) {
// CHECK-NEXT:   return max(f, g) + min(f, g);
// CHECK-NEXT: }
__device__ float __host__ foo3(float f, float g) {
  return max(f, g) + min(f, g);
}

typedef int INT;
typedef unsigned UINT;
using int_t = int;
using uint_t = unsigned;

// CHECK: int foo(UINT i, INT j) {
// CHECK-NEXT:   return dpct::max(i, j) + dpct::min(i, j);
// CHECK-NEXT: }
int foo(UINT i, INT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int foo(INT i, UINT j) {
// CHECK-NEXT:   return dpct::max(i, j) + dpct::min(i, j);
// CHECK-NEXT: }
int foo(INT i, UINT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(uint_t i, int_t j) {
// CHECK-NEXT:   return dpct::max(i, j) + dpct::min(i, j);
// CHECK-NEXT: }
int bar(uint_t i, int_t j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(int_t i, uint_t j) {
// CHECK-NEXT:   return dpct::max(i, j) + dpct::min(i, j);
// CHECK-NEXT: }
int bar(int_t i, uint_t j) {
  return max(i, j) + min(i, j);
}

__device__ void test_pow() {
  int i;
  float f;
  double d;

  // CHECK: dpct::pow(i, i);
  pow(i, i);
  // CHECK: dpct::pow(f, i);
  pow(f, i);
  // CHECK: dpct::pow(d, i);
  pow(d, i);

  // CHECK: dpct::pow(i, f);
  pow(i, f);
  // CHECK: dpct::pow(f, f);
  pow(f, f);
  // CHECK: dpct::pow(d, f);
  pow(d, f);

  // CHECK: dpct::pow(i, d);
  pow(i, d);
  // CHECK: dpct::pow(f, d);
  pow(f, d);
  // CHECK: dpct::pow(d, d);
  pow(d, d);
}

__global__ void foobar(int i) {
  // CHECK: dpct::max(i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_id(2), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_id(1), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_id(0), i);
  max(i, threadIdx.x);
  max(i, threadIdx.y);
  max(i, threadIdx.z);
  max(threadIdx.x, i);
  max(threadIdx.y, i);
  max(threadIdx.z, i);

  // CHECK: dpct::max(i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_group(2), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_group(1), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_group(0), i);
  max(i, blockIdx.x);
  max(i, blockIdx.y);
  max(i, blockIdx.z);
  max(blockIdx.x, i);
  max(blockIdx.y, i);
  max(blockIdx.z, i);

  // CHECK: dpct::max(i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: dpct::max(i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_range(2), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_range(1), i);
  // CHECK-NEXT: dpct::max((unsigned int)item_ct1.get_local_range(0), i);
  max(i, blockDim.x);
  max(i, blockDim.y);
  max(i, blockDim.z);
  max(blockDim.x, i);
  max(blockDim.y, i);
  max(blockDim.z, i);

  // CHECK: dpct::min(i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_id(2), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_id(1), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_id(0), i);
  min(i, threadIdx.x);
  min(i, threadIdx.y);
  min(i, threadIdx.z);
  min(threadIdx.x, i);
  min(threadIdx.y, i);
  min(threadIdx.z, i);

  // CHECK: dpct::min(i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_group(2), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_group(1), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_group(0), i);
  min(i, blockIdx.x);
  min(i, blockIdx.y);
  min(i, blockIdx.z);
  min(blockIdx.x, i);
  min(blockIdx.y, i);
  min(blockIdx.z, i);

  // CHECK: dpct::min(i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: dpct::min(i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_range(2), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_range(1), i);
  // CHECK-NEXT: dpct::min((unsigned int)item_ct1.get_local_range(0), i);
  min(i, blockDim.x);
  min(i, blockDim.y);
  min(i, blockDim.z);
  min(blockDim.x, i);
  min(blockDim.y, i);
  min(blockDim.z, i);
}

void do_migration() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}
__global__ void do_migration2() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__device__ void do_migration3() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__host__ __device__ void do_migration4() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}
namespace t {
int max(int i, int j) {
  return i > j ? i : j;
}
}
void do_migration8() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}
void no_migration2() {
  int i, j;
  // CHECK: t::max(i, j);
  t::max(i, j);
}
void no_migration3() {
  int i, j;
  // CHECK: std::max(i, j);
  std::max(i, j);
}
__host__ void do_migration9() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}

void ns() {
  using namespace std;
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}

void no_migration5() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: std::fabs(f);
  //CHECK-NEXT: std::nearbyintf(f);
  //CHECK-NEXT: std::remquof(f, f, &i);
  //CHECK-NEXT: std::acoshf(f);
  //CHECK-NEXT: std::asinhf(f);
  //CHECK-NEXT: std::abs(f);
  //CHECK-NEXT: std::frexp(f, &i);
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: std::nearbyint(f);
  //CHECK-NEXT: std::remquo(f, f, &i);
  //CHECK-NEXT: std::acos(f);
  //CHECK-NEXT: std::acosh(f);
  //CHECK-NEXT: std::asin(f);
  //CHECK-NEXT: std::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acoshf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);

  int64_t a;
  //CHECK: std::max<int64_t>(a, 1);
  std::max<int64_t>(a, 1);
}

__device__ void do_migration5() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acoshf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__global__ void do_migration6() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acoshf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ __host__ void do_migration7() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, int>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acoshf(f);
  std::asinhf(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ void test_recursive_unary() {
  int i, j, k;
  // CHECK: sycl::max(-sycl::max(-sycl::abs(i), j), k);
  max(-max(-abs(i), j), k);
}

__device__ void do_math(int i, int j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt((double)i);
  sqrt(i);
  // CHECK: sycl::fmod((double)i, (double)j);
  fmod(i, j);
  // CHECK: sycl::sin((double)i);
  sin(i);
  // CHECK: sycl::cos((double)i);
  cos(i);
}

__device__ void do_math(float i, float j) {
  // CHECK: sycl::sqrt(i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}

__device__ void do_math(double i, double j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}

__global__ void k() {
  float f;

  char c;
  unsigned char uc;
  short s;
  unsigned short us;
  int i;
  unsigned int ui;
  long l;
  unsigned long ul;
  long long ll;
  unsigned long long ull;

  // CHECK: f * f;
  pow(f, 2);
  // CHECK: dpct::pow(f, 3);
  pow(f, 3);
  // CHECK: f * f;
  powf(f, 2);
  // CHECK: dpct::pow(f, 3);
  powf(f, 3);
  // CHECK: f * f;
  __powf(f, 2);
  // CHECK: dpct::pow(f, 3);
  __powf(f, 3);

  // CHECK: dpct::pow(f, c);
  pow(f, c);
  // CHECK: dpct::pow(f, uc);
  pow(f, uc);
  // CHECK: dpct::pow(f, s);
  pow(f, s);
  // CHECK: dpct::pow(f, us);
  pow(f, us);
  // CHECK: dpct::pow(f, i);
  pow(f, i);
  // CHECK: dpct::pow(f, ui);
  pow(f, ui);
  // CHECK: dpct::pow(f, l);
  pow(f, l);
  // CHECK: dpct::pow(f, ul);
  pow(f, ul);
  // CHECK: dpct::pow(f, ll);
  pow(f, ll);
  // CHECK: dpct::pow(f, ull);
  pow(f, ull);

  // CHECK: dpct::pow(f, c);
  powf(f, c);
  // CHECK: dpct::pow(f, uc);
  powf(f, uc);
  // CHECK: dpct::pow(f, s);
  powf(f, s);
  // CHECK: dpct::pow(f, us);
  powf(f, us);
  // CHECK: dpct::pow(f, i);
  powf(f, i);
  // CHECK: dpct::pow(f, ui);
  powf(f, ui);
  // CHECK: dpct::pow(f, l);
  powf(f, l);
  // CHECK: dpct::pow(f, ul);
  powf(f, ul);
  // CHECK: dpct::pow(f, ll);
  powf(f, ll);
  // CHECK: dpct::pow(f, ull);
  powf(f, ull);

  // CHECK: dpct::pow(f, c);
  __powf(f, c);
  // CHECK: dpct::pow(f, uc);
  __powf(f, uc);
  // CHECK: dpct::pow(f, s);
  __powf(f, s);
  // CHECK: dpct::pow(f, us);
  __powf(f, us);
  // CHECK: dpct::pow(f, i);
  __powf(f, i);
  // CHECK: dpct::pow(f, ui);
  __powf(f, ui);
  // CHECK: dpct::pow(f, l);
  __powf(f, l);
  // CHECK: dpct::pow(f, ul);
  __powf(f, ul);
  // CHECK: dpct::pow(f, ll);
  __powf(f, ll);
  // CHECK: dpct::pow(f, ull);
  __powf(f, ull);

}

__global__ void k2() {
  int i, i2;
  unsigned u, u1, u2;
  float f0, f1, f2, f3;
  double d0, d1, d2, d3;
  long l, l2;
  long long ll, ll2;
  unsigned long long ull, ull2;

  // CHECK: sycl::exp(d0*d0)*sycl::erfc(d0);
  erfcx(d0);
  // CHECK: sycl::exp(f0*f0)*sycl::erfc(f0);
  erfcxf(f0);
  // CHECK: sycl::length(sycl::double3(d0, d1, d2));
  norm3d(d0, d1, d2);
  // CHECK: sycl::length(sycl::float3(f0, f1, f2));
  norm3df(f0, f1, f2);
  // CHECK: sycl::length(sycl::double4(d0, d1, d2, d3));
  norm4d(d0, d1, d2, d3);
  // CHECK: sycl::length(sycl::float4(f0, f1, f2, f3));
  norm4df(f0, f1, f2, f3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::cbrt call is used instead of the rcbrt call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: 1 / dpct::cbrt<double>(d0);
  rcbrt(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::cbrt call is used instead of the rcbrtf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: sycl::native::recip(dpct::cbrt<float>(f0));
  rcbrtf(f0);
  // CHECK: 1 / sycl::length(sycl::double3(d0, d1, d2));
  rnorm3d(d0, d1, d2);
  // CHECK: sycl::native::recip(sycl::length(sycl::float3(f0, f1, f2)));
  rnorm3df(f0, f1, f2);
  // CHECK: 1 / sycl::length(sycl::double4(d0, d1, d2, d3));
  rnorm4d(d0, d1, d2, d3);
  // CHECK: sycl::native::recip(sycl::length(sycl::float4(f0, f1, f2, f3)));
  rnorm4df(f0, f1, f2, f3);
  // CHECK: d0*(2<<l);
  scalbln(d0, l);
  // CHECK: f0*(2<<l);
  scalblnf(f0, l);
  // CHECK: d0*(2<<i);
  scalbn(d0, i);
  // CHECK: f0*(2<<i);
  scalbnf(f0, i);
  // CHECK: dpct::cast_double_to_int(d0);
  __double2hiint(d0);
  // CHECK: dpct::cast_double_to_int(d0, false);
  __double2loint(d0);
  // CHECK: dpct::cast_ints_to_double(i, i2);
  __hiloint2double(i, i2);

  // CHECK: sycl::abs_diff(i, i2)+u;
  __sad(i, i2, u);
  // CHECK: sycl::abs_diff(u, u1)+u2;
  __usad(u, u1, u2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rn(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rz(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/(d0+d0));
  __drcp_rz(d0+d0);

  // CHECK: sycl::mul_hi(ll, ll);
  __mul64hi(ll, ll);
  // CHECK: sycl::rhadd(i, i2);
  __rhadd(i, i2);
  // CHECK: sycl::hadd(u, u2);
  __uhadd(u, u2);
  // CHECK: sycl::mul24(u, u2);
  __umul24(u, u2);
  // CHECK: sycl::mul_hi(ull, ull2);
  __umul64hi(ull, ull2);
  // CHECK: sycl::mul_hi(u, u2);
  __umulhi(u, u2);
  // CHECK: sycl::rhadd(u, u2);
  __urhadd(u, u2);

  double *a_d;
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 0);
  norm(0, a_d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 1);
  norm(1, a_d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 2);
  norm(2, a_d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 3);
  norm(3, a_d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 4);
  norm(4, a_d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The dpct::length call is used instead of the norm call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::length(a_d, 5);
  norm(5, a_d);
}

// CHECK: #define MUL(a, b) sycl::mul24((int)a, (int)b)
#define MUL(a, b) __mul24(a, b)
__global__ void test_mul24_complicated() {
  // CHECK: unsigned int      tid = sycl::mul24((int)item_ct1.get_local_range(2), (int)item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int      tid = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int  threadN = sycl::mul24((int)item_ct1.get_local_range(2), (int)item_ct1.get_group_range(2));
  unsigned int  threadN = __mul24(blockDim.x, gridDim.x);

  // CHECK: unsigned int     tid2 = MUL(item_ct1.get_local_range(2), item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int     tid2 = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int threadN2 = MUL(item_ct1.get_local_range(2), item_ct1.get_group_range(2));
  unsigned int threadN2 = MUL(blockDim.x, gridDim.x);
}

// CHECK: #define UMUL(a, b) sycl::mul24((unsigned int)a, (unsigned int)b)
#define UMUL(a, b) __umul24(a, b)

__global__ void test_umul24_complicated() {
  // CHECK: unsigned int      tid = sycl::mul24((unsigned int)item_ct1.get_local_range(2), (unsigned int)item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int      tid = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int  threadN = sycl::mul24((unsigned int)item_ct1.get_local_range(2), (unsigned int)item_ct1.get_group_range(2));
  unsigned int  threadN = __umul24(blockDim.x, gridDim.x);

  // CHECK: unsigned int     tid2 = UMUL(item_ct1.get_local_range(2), item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int     tid2 = UMUL(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int threadN2 = UMUL(item_ct1.get_local_range(2), item_ct1.get_group_range(2));
  unsigned int threadN2 = UMUL(blockDim.x, gridDim.x);
}

struct S {
  int m;
};

__device__ int fun(int i) { return i * 2; }

__device__ S fun2(int i) { return { i * 2 }; }

#define TWO 2.0

// CHECK: #define POW_TWO(x) dpct::pow(x, 2.0)
#define POW_TWO(x) pow(x, 2.0)

// CHECK: #define POW(x, y) dpct::pow(x, y)
#define POW(x, y) pow(x, y)

__global__ void test_side_effects() {
  int a, b[10];
  S s;
  S *sp = new S;

  // CHECK: int c = (a - b[0]) * (a - b[0]);
  int c = pow(a - b[0], 2);
  // CHECK: int d = a * a;
  int d = pow(a, 2);
  // CHECK: int e = 2 * 2;
  int e = pow(2, 2);
  // CHECK: int f = 2.0 * 2.0;
  int f = pow(2.0, 2);
  // CHECK: int g = (a ? b[0] : b[1]) * (a ? b[0] : b[1]);
  int g = pow(a ? b[0] : b[1] , 2);
  // CHECK: int h = (a >> 2) * (a >> 2);
  int h = pow(a >> 2, 2);
  // CHECK: int i = dpct::pow(fun(a), 2);
  int i = pow(fun(a), 2);
  // CHECK: int j = b[0] * b[0];
  int j = pow(b[0], 2);
  // CHECK: int k = (a + b[0]) * (a + b[0]);
  int k = pow((a + b[0]), 2);
  // CHECK: int l = s.m * s.m;
  int l = pow(s.m, 2);
  // CHECK: int m = sp->m * sp->m;
  int m = pow(sp->m, 2);
  // CHECK: int n = (int)a * (int)a;
  int n = pow((int)a, 2);
  // CHECK: int o = static_cast<float>(a) * static_cast<float>(a);
  int o = pow(static_cast<float>(a), 2);
  // CHECK: int p = (a & b[0]) * (a & b[0]);
  int p = pow(a & b[0], 2);
  // CHECK: int q = (a && b[0]) * (a && b[0]);
  int q = pow(a && b[0], 2);
  // CHECK: int r = dpct::pow(a += b[0], 2);
  int r = pow(a += b[0], 2);
  // CHECK: int t = dpct::pow(fun2(a).m, 2);
  int t = pow(fun2(a).m, 2);
  // CHECK: int u = dpct::pow(a = b[0], 2);
  int u = pow(a = b[0], 2);

  // CHECK: int u1 = 2.0 * 2.0;
  int u1 = pow(2.0, 2.0);
  // CHECK: int v = 2.0 * 2.0;
  int v = pow(2.0, 1.99999999999999999);
  // CHECK: int w = 2.0 * 2.0;
  int w = pow(2.0, 2.0f);
  // CHECK: int w1 = 2.0 * 2.0;
  int w1 = pow(2.0, 2.0000000001f);
  // CHECK: int w2 = 2.0 * 2.0;
  int w2 = pow(2.0, 2.0000001f);
  // CHECK: int w3 = 2.0 * 2.0;
  int w3 = pow(2.0, 2.0000000000000001);
  // CHECK: int x = 2.0 * 2.0;
  int x = pow(2.0, 2l);
  // CHECK: int y = 2.0 * 2.0;
  int y = pow(2.0, 2ul);
  // CHECK: int z = 2.0 * 2.0;
  int z = pow(2.0, 2ull);

  // CHECK: dpct::pow(2.0, TWO);
  pow(2.0, TWO);
  // CHECK: POW_TWO(2.0);
  POW_TWO(2.0);
  // CHECK: POW(2.0, 2.0);
  POW(2.0, 2.0);
}

#define fp float
__device__ void foo() {
  fp d_initvalu_36;
  fp ret;
  // CHECK: ret = dpct::pow(d_initvalu_36, fp(1.6));
  ret = pow(d_initvalu_36, fp(1.6));
}

// CHECK: template <typename... T>
// CHECK-NEXT: static void log(T... o)
// CHECK-NEXT: {
// CHECK-NEXT:   std::cout << "log" << std::endl;
// CHECK-NEXT: }
template <typename... T>
static void log(T... o)
{
  std::cout << "log" << std::endl;
}

// CHECK: template <typename... T>
// CHECK-NEXT: void log_log(bool cond, T... o)
// CHECK-NEXT: {
// CHECK-NEXT:   if (cond)
// CHECK-NEXT:   {
// CHECK-NEXT:     log(o...);
// CHECK-NEXT:   }
// CHECK-NEXT: }
template <typename... T>
void log_log(bool cond, T... o)
{
  if (cond)
  {
    log(o...);
  }
}

// CHECK: int test_log() {
// CHECK-NEXT:   log_log(true, "MJ", "23");
// CHECK-NEXT: }
int test_log() {
  log_log(true, "MJ", "23");
}

__device__ void bar1(double *d) {
  int i;
  double d1;
  double &d2 = *d;

  // CHECK: DPCT1081:{{[0-9]+}}: The generated code assumes that "&d2" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  // CHECK: d1 = sycl::sincos((double)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(&d2));
  sincos(i, &d1, &d2);
}

__device__ void bar1(double *d, bool flag) {
  int i;
  double d1;
  double d2;
  double *d2_p;
  d2_p = &d2;

  //CHECK:d1 = sycl::sincos((double)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, double>(d2_p));
  //CHECK-NEXT:if (flag) {
  //CHECK-NEXT:  d2_p = d + 2;
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1081:{{[0-9]+}}: The generated code assumes that "d2_p" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  //CHECK-NEXT:*/
  //CHECK-NEXT:d1 = sycl::sincos((double)i, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, double>(d2_p));
  //CHECK-NEXT:d2_p = &d2;
  //CHECK-NEXT:d1 = sycl::sincos((double)i, sycl::address_space_cast<sycl::access::address_space::private_space, sycl::access::decorated::yes, double>(d2_p));
  sincos(i, &d1, d2_p);
  if (flag) {
    d2_p = d + 2;
  }
  sincos(i, &d1, d2_p);
  d2_p = &d2;
  sincos(i, &d1, d2_p);
}

__device__ int* get_ptr();
__device__ void bar2() {
  double d0;
  int i;
  //CHECK:/*
  //CHECK-NEXT:DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1081:{{[0-9]+}}: The generated code assumes that "get_ptr() + i" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  //CHECK-NEXT:*/
  //CHECK-NEXT:double d2 = sycl::frexp(d0, sycl::address_space_cast<sycl::access::address_space::global_space, sycl::access::decorated::yes, int>(get_ptr() + i));
  double d2 = frexp(d0, get_ptr() + i);
}

__device__ void foo_lambda1()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = std::min(x, 10) + std::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

__device__ __host__ void foo_lambda2()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = std::min(x, 10) + std::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

__global__ void foo_lambda3()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = std::min(x, 10) + std::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

void foo_lambda4()
{
  []()
  {
    int num = 256;
    // CHECK: auto x = std::min<long long>(num, 10);
    auto x = std::min<long long>(num, 10);
    // CHECK: auto y = std::max<float>(100.0f, num);
    auto y = std::max<float>(100.0f, num);
  }();
}

void foo_lambda5()
{
  auto foo = []()
  {
    int num = 256;
    // CHECK: auto x = std::min<long long>(num, 10);
    auto x = std::min<long long>(num, 10);
    // CHECK: auto y = std::max<float>(100.0f, num);
    auto y = std::max<float>(100.0f, num);
  };
  foo();
}

void foo_lambda6()
{
  []()
  {
    []()
    {
      int num = 256;
      // CHECK: auto x = std::min<long long>(num, 10);
      auto x = std::min<long long>(num, 10);
      // CHECK: auto y = std::max<float>(100.0f, num);
      auto y = std::max<float>(100.0f, num);
    }();
  }();
}

auto static_foo = []()
{
  int num = 256;
  // CHECK: auto x = std::min(num, 10);
  auto x = std::min(num, 10);
  // CHECK: auto y = std::max(100, num);
  auto y = std::max(100, num);
};
void foo_lambda7()
{
  static_foo();
}

__device__ void qualified(double d1,
                          const double d2,
			  volatile double d3,
			  const volatile double d4) {
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rd(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rd(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rd(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rd(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rn(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rn(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rn(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rn(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_ru(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_ru(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_ru(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_ru(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rz(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rz(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rz(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different accuracy than the original code. Verify the correctness. SYCL math built-in function rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rz(d4);
}

__global__ void foo4(unsigned char *uc, int i) {
  float f0, f1;
  int tid = blockDim.x * blockIdx.x + threadIdx.x + i + 1;
  // CHECK: uc[tid] = sycl::sqrt(f0 * f0 + f1 * f1);
  uc[tid] = sqrtf(powf(f0, 2.f) + powf(f1, 2.f));
}

void foo5() {
  double d0;
  float f0;
  int i;

  // CHECK: d0 = ceil(d0);
  d0 = ceil(d0);
  // CHECK: d0 = ceil(i);
  d0 = ceil(i);

  // CHECK: f0 = ceilf(f0);
  f0 = ceilf(f0);
  // CHECK: f0 = ceilf(i);
  f0 = ceilf(i);
}
