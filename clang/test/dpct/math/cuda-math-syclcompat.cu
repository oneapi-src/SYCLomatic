// RUN: dpct --format-range=none -out-root %T/math/cuda-math-syclcompat %s -use-syclcompat --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/math/cuda-math-syclcompat/cuda-math-syclcompat.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/math/cuda-math-syclcompat/cuda-math-syclcompat.dp.cpp -o %T/math/cuda-math-syclcompat/cuda-math-syclcompat.dp.o %}


#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void kernelFuncBfloat162Arithmetic() {
  __hip_bfloat162 bf162, bf162_1, bf162_2, bf162_3;
  // CHECK: bf162 = syclcompat::cmul_add(bf162_1, bf162_2, bf162_3);
  bf162 = __hcmadd(bf162_1, bf162_2, bf162_3);

  __half2 h2, h2_1, h2_2;
  // CHECK: h2_2 = syclcompat::cmul_add(h2, h2_1, h2_2);
  h2_2 = __hcmadd(h2, h2_1, h2_2);
}