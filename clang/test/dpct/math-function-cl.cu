// RUN: dpct --format-range=none --out-root %T/math-function-cl %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/math-function-cl/math-function-cl.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/math-function-cl/math-function-cl.dp.cpp -o %T/math-function-cl/math-function-cl.dp.o %}


#include <hip/hip_runtime.h>
#include <cmath>

#include <math_functions.h>

__device__ float4 fun() {
  float4 a, b, c;
  // CHECK: sycl::fma(a.x(), b.x(), c.x());
  __fmaf_rn(a.x, b.x, c.x);
  // CHECK: return sycl::float4(sycl::fma(a.x(), b.x(), c.x()), sycl::fma(a.y(), b.y(), c.y()), sycl::fma(a.z(), b.z(), c.z()), sycl::fma(a.w(), b.w(), c.w()));
  return make_float4(__fmaf_rd(a.x, b.x, c.x), __fmaf_rz(a.y, b.y, c.y), __fmaf_rn(a.z, b.z, c.z), __fmaf_rn(a.w, b.w, c.w));
}


__global__ void kernel() {

}

void foo() {
  // CHECK:   dpct::get_in_order_queue().parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, ceil(2.3)), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {
  // CHECK-NEXT:           kernel();
  // CHECK-NEXT:         });
  kernel<<< ceil(2.3), 1 >>>();
}

int main() {
  return 0;
}
