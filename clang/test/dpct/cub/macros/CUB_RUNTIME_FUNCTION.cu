// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/type/HIPCUB_RUNTIME_FUNCTION %S/HIPCUB_RUNTIME_FUNCTION.cu --cuda-include-path="%cuda-path/include" -- -std=c++17 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/type/HIPCUB_RUNTIME_FUNCTION/HIPCUB_RUNTIME_FUNCTION.dp.cpp %s

#include <hipcub/hipcub.hpp>

// CHECK: void foo() {}
HIPCUB_RUNTIME_FUNCTION void foo() {}
