#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// UNSUPPORTED: system-windows
// RUN: dpct -in-root %S -out-root %T/blocklevel/blockstore %S/blockstore.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockstore/blockstore.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/blocklevel/blockstore/blockstore.dp.cpp -o %T/blocklevel/blockstore/blockstore.dp.o %}

#include <hipcub/hipcub.hpp>

__global__ void BlockedKernel(int *d_data, int valid_items) {
  // Specialize BlockStore for a 1D block of 128 threads owning 4 integer items each
  // CHECK: using BlockStore = dpct::group::group_store<int, 4>;
  using BlockStore = hipcub::BlockStore<int, 128, 4>;

  __shared__ typename BlockStore::TempStorage temp_storage;

  int thread_data[4];
  thread_data[0] = threadIdx.x * 4 + 0;
  thread_data[1] = threadIdx.x * 4 + 1;
  thread_data[2] = threadIdx.x * 4 + 2;
  thread_data[3] = threadIdx.x * 4 + 3;

  // CHECK: BlockStore(temp_storage).store(item_ct1, d_data, thread_data, valid_items);
  BlockStore(temp_storage).Store(d_data, thread_data, valid_items);
}

__global__ void StripedKernel(int *d_data, int valid_items) {
  // Specialize BlockStore for a 1D block of 128 threads owning 4 integer items each
  // CHECK: using BlockStore = dpct::group::group_store<int, 4, dpct::group::group_store_algorithm::striped>;
  using BlockStore = hipcub::BlockStore<int, 128, 4, hipcub::BLOCK_STORE_STRIPED>;

  __shared__ typename BlockStore::TempStorage temp_storage;

  int thread_data[4];
  thread_data[0] = threadIdx.x * 4 + 0;
  thread_data[1] = threadIdx.x * 4 + 1;
  thread_data[2] = threadIdx.x * 4 + 2;
  thread_data[3] = threadIdx.x * 4 + 3;
  // CHECK: BlockStore(temp_storage).store(item_ct1, d_data, thread_data, valid_items);
  BlockStore(temp_storage).Store(d_data, thread_data, valid_items);
}

int main() {
  int *d_data;
  hipMallocManaged(&d_data, sizeof(int) * 512);
  hipMemset(d_data, 0, sizeof(int) * 512);
  // CHECK: q_ct1.submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     sycl::local_accessor<uint8_t, 1> temp_storage_acc(dpct::group::group_store<int, 4>::get_local_memory_size(sycl::range<3>(1, 1, 128).size()), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 128), sycl::range<3>(1, 1, 128)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         BlockedKernel(d_data, 5, item_ct1, &temp_storage_acc[0]);
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  BlockedKernel<<<1, 128>>>(d_data, 5);
  hipStreamSynchronize(0);
  for (int i = 0; i < 512; ++i)
    printf("%d%c", d_data[i], (i == 511 ? '\n' : ' '));
  hipMemset(d_data, 0, sizeof(int) * 512);
  // CHECK: q_ct1.submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     sycl::local_accessor<uint8_t, 1> temp_storage_acc(dpct::group::group_store<int, 4>::get_local_memory_size(sycl::range<3>(1, 1, 128).size()), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 128), sycl::range<3>(1, 1, 128)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         StripedKernel(d_data, 5, item_ct1, &temp_storage_acc[0]);
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  StripedKernel<<<1, 128>>>(d_data, 5);
  hipStreamSynchronize(0);
  for (int i = 0; i < 512; ++i)
    printf("%d%c", d_data[i], (i == 511 ? '\n' : ' '));
  return 0;
}
