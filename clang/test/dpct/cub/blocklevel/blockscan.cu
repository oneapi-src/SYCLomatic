#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/blocklevel/blockscan %S/blockscan.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockscan/blockscan.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/blocklevel/blockscan/blockscan.dp.cpp -o %T/blocklevel/blockscan/blockscan.dp.o %}

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define WARP_SIZE 32

void init_data(int* data, int num) {
  for(int i = 0; i < num; i++)
    data[i] = 1;
}
void verify_data(int* data, int num) {
  return;
}
void print_data(int* data, int num) {
  for (int i = 0; i < num; i++) {
    std::cout << data[i] << " ";
  }
  std::cout << std::endl;
}
//CHECK: void ExclusiveScanKernel(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::exclusive_scan_over_group(item_ct1.get_group(), input, 0, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void ExclusiveScanKernel(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).ExclusiveScan(input, output, 0, hipcub::Sum());
  data[threadid] = output;
}

//CHECK: void ExclusiveScanKernel_Max(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::exclusive_scan_over_group(item_ct1.get_group(), input, 0, sycl::maximum<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void ExclusiveScanKernel_Max(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).ExclusiveScan(input, output, 0, hipcub::Max());
  data[threadid] = output;
}

//CHECK: void ExclusiveScanKernel_Min(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::exclusive_scan_over_group(item_ct1.get_group(), input, 0, sycl::minimum<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void ExclusiveScanKernel_Min(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).ExclusiveScan(input, output, 0, hipcub::Min());
  data[threadid] = output;
}

//CHECK: void ExclusiveSumKernel(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::exclusive_scan_over_group(item_ct1.get_group(), input, 0, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void ExclusiveSumKernel(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).ExclusiveSum(input, output);
  data[threadid] = output;
}

//CHECK: void InclusiveScanKernel(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::inclusive_scan_over_group(item_ct1.get_group(), input, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void InclusiveScanKernel(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).InclusiveScan(input, output, hipcub::Sum());
  data[threadid] = output;
}

//CHECK: void InclusiveScanKernel_Max(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::inclusive_scan_over_group(item_ct1.get_group(), input, sycl::maximum<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void InclusiveScanKernel_Max(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).InclusiveScan(input, output, hipcub::Max());
  data[threadid] = output;
}

//CHECK: void InclusiveScanKernel_Min(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::inclusive_scan_over_group(item_ct1.get_group(), input, sycl::minimum<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void InclusiveScanKernel_Min(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).InclusiveScan(input, output, hipcub::Min());
  data[threadid] = output;
}

//CHECK: void InclusiveSumKernel(int* data,
//CHECK-NEXT:   const sycl::nd_item<3> &item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::inclusive_scan_over_group(item_ct1.get_group(), input, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void InclusiveSumKernel(int* data) {
  typedef hipcub::BlockScan<int, 4> BlockScan;

  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).InclusiveSum(input, output);
  data[threadid] = output;
}

// CHECK: template <int THREADS_PER_BLOCK>
// CHECK: void SomeKernel(int *data,
// CHECK：                const sycl::nd_item<3> &item_ct1) {
// CHECK-NOT: typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan; 
// CHECK:  int threadid = item_ct1.get_local_id(2);
// CHECK:  int input = data[threadid];
// CHECK:  int output = 0;
// CHECK:  output = sycl::inclusive_scan_over_group(item_ct1.get_group(), input, sycl::plus<>());
// CHECK:  data[threadid] = output;
// CHECK:}
template <int THREADS_PER_BLOCK>
__global__ void SomeKernel(int *data) {
   typedef hipcub::BlockScan<int, THREADS_PER_BLOCK> BlockScan;
  __shared__ typename BlockScan::TempStorage temp1;
  int threadid = threadIdx.x;
  int input = data[threadid];
  int output = 0;
  BlockScan(temp1).InclusiveSum(input, output);
  data[threadid] = output;
}

int main() {
  int* dev_data = nullptr;

  dim3 GridSize(2);
  dim3 BlockSize(1 , 1, 128);
  int TotalThread = GridSize.x * BlockSize.x * BlockSize.y * BlockSize.z;

  hipMallocManaged(&dev_data, TotalThread * sizeof(int));

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          ExclusiveScanKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  ExclusiveScanKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          ExclusiveScanKernel_Max(dev_data, item_ct1);
//CHECK-NEXT:        });
  ExclusiveScanKernel_Max<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          ExclusiveScanKernel_Min(dev_data, item_ct1);
//CHECK-NEXT:        });
  ExclusiveScanKernel_Min<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          ExclusiveSumKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  ExclusiveSumKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          InclusiveScanKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  InclusiveScanKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          InclusiveScanKernel_Max(dev_data, item_ct1);
//CHECK-NEXT:        });
  InclusiveScanKernel_Max<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          InclusiveScanKernel_Min(dev_data, item_ct1);
//CHECK-NEXT:        });
  InclusiveScanKernel_Min<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          InclusiveSumKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  InclusiveSumKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

// CHECK: q_ct1.parallel_for(
// CHECK-NEXT: sycl::nd_range<3>(GridSize * BlockSize, BlockSize), 
// CHECK-NEXT: [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT: SomeKernel<4>(dev_data, item_ct1);
// CHECK-NEXT: });
  SomeKernel<4><<<GridSize, BlockSize>>>(dev_data);

  return 0;
}
