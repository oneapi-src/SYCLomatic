#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/blocklevel/blockscan_p2 %S/blockscan_p2.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockscan_p2/blockscan_p2.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/blocklevel/blockscan_p2/blockscan_p2.dp.cpp -o %T/blocklevel/blockscan_p2/blockscan_p2.dp.o %}

#include <iostream>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define WARP_SIZE 32
#define DATA_NUM 256


template<typename T = int>
void init_data(T* data, int num) {
  for(int i = 0; i < num; i++)
    data[i] = i;
}

template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  for(int i = 0; i < num; i = i + step) {
    if(data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

template<typename T = int>
void print_data(T* data, int num) {
  for (int i = 0; i < num; i++) {
    std::cout << data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

//CHECK: void ExclusiveSumKernel1(int* data, int* aggregate,
//CHECK-NEXT:                          const sycl::nd_item<3> &item_ct1) {
//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                        + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                        + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                        + item_ct1.get_local_id(2);;
//CHECK:    int input = data[threadid];
//CHECK:    int output = 0;
//CHECK:    int agg = 0;

//CHECK:    output = dpct::group::exclusive_scan(item_ct1, input, 0, sycl::plus<>(), agg);

//CHECK:    data[threadid] = output;
//CHECK:    aggregate[threadid] = agg;
//CHECK:  }
__global__ void ExclusiveSumKernel1(int* data, int* aggregate) {
  typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
  __shared__ typename BlockScan::TempStorage temp1;

  int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
  int input = data[threadid];
  int output = 0;
  int agg = 0;

  BlockScan(temp1).ExclusiveSum(input, output, agg);

  data[threadid] = output;
  aggregate[threadid] = agg;
}

//CHECK:  struct CallbackOp1
//CHECK:  {
//CHECK:      int value;
//CHECK:      CallbackOp1(int init_value) : value(init_value) {}
//CHECK:      int operator()(int aggregate)
//CHECK:      {
//CHECK:          int pre_value = value;
//CHECK:          value += aggregate;
//CHECK:          return pre_value;
//CHECK:      }
//CHECK:  };

//CHECK:  void ExclusiveSumKernel2(int* data, const sycl::nd_item<3> &item_ct1) {

//CHECK:      CallbackOp1 CB(0);
//CHECK:      int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                        + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                        + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                        + item_ct1.get_local_id(2);;
//CHECK:      int input = data[threadid];
//CHECK:      int output = 0;

//CHECK:      output = dpct::group::exclusive_scan(item_ct1, input, sycl::plus<>(), CB);

//CHECK:      data[threadid] = output;
//CHECK:  }

struct CallbackOp1
{
    int value;

    __device__ CallbackOp1(int init_value) : value(init_value) {}

    __device__ int operator()(int aggregate)
    {
        int pre_value = value;
        value += aggregate;
        return pre_value;
    }
};

__global__ void ExclusiveSumKernel2(int* data) {
    typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    CallbackOp1 CB(0);
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input = data[threadid];
    int output = 0;

    BlockScan(temp1).ExclusiveSum(input, output, CB);

    data[threadid] = output;
}
//CHECK:  void ExclusiveSumKernel3(int* data, const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:    int input[4];
//CHECK:    input[0] = data[4 * threadid];
//CHECK:    input[1] = data[4 * threadid + 1];
//CHECK:    input[2] = data[4 * threadid + 2];
//CHECK:    input[3] = data[4 * threadid + 3];
//CHECK:    int output[4];

//CHECK:    dpct::group::exclusive_scan(item_ct1, input, output, 0, sycl::plus<>());

//CHECK:    data[4 * threadid] = output[0];
//CHECK:    data[4 * threadid + 1] = output[1];
//CHECK:    data[4 * threadid + 2] = output[2];
//CHECK:    data[4 * threadid + 3] = output[3];
//CHECK:  }
__global__ void ExclusiveSumKernel3(int* data) {
    typedef hipcub::BlockScan<int, 8, hipcub::BLOCK_SCAN_RAKING, 4, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input[4];
    input[0] = data[4 * threadid];
    input[1] = data[4 * threadid + 1];
    input[2] = data[4 * threadid + 2];
    input[3] = data[4 * threadid + 3];
    int output[4];

    BlockScan(temp1).ExclusiveSum(input, output);

    data[4 * threadid] = output[0];
    data[4 * threadid + 1] = output[1];
    data[4 * threadid + 2] = output[2];
    data[4 * threadid + 3] = output[3];
}
//CHECK:  void ExclusiveScanKernel1(int* data, int* aggregate,
//CHECK: const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:  int input = data[threadid];
//CHECK:  int output = 0;
//CHECK:  int agg = 0;

//CHECK:  output = dpct::group::exclusive_scan(item_ct1, input, 0, sycl::plus<>(), agg);

//CHECK:  data[threadid] = output;
//CHECK:  aggregate[threadid] = agg;
//CHECK:  }
__global__ void ExclusiveScanKernel1(int* data, int* aggregate) {
  typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
  __shared__ typename BlockScan::TempStorage temp1;

    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
  int input = data[threadid];
  int output = 0;
  int agg = 0;

  BlockScan(temp1).ExclusiveScan(input, output, 0, hipcub::Sum(), agg);

  data[threadid] = output;
  aggregate[threadid] = agg;
}

//CHECK:  struct CallbackOp2
//CHECK:  {
//CHECK:      int value;
//CHECK:      CallbackOp2(int init_value) : value(init_value) {}
//CHECK:      int operator()(int aggregate)
//CHECK:      {
//CHECK:          int pre_value = value;
//CHECK:          value += aggregate;
//CHECK:          return pre_value;
//CHECK:      }
//CHECK:  };

//CHECK:  void ExclusiveScanKernel2(int* data, const sycl::nd_item<3> &item_ct1) {

//CHECK:      CallbackOp2 CB(0);
//CHECK:      int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                        + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                        + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                        + item_ct1.get_local_id(2);;
//CHECK:      int input = data[threadid];
//CHECK:      int output = 0;

//CHECK:      output = dpct::group::exclusive_scan(item_ct1, input, sycl::plus<>(), CB);

//CHECK:      data[threadid] = output;
//CHECK:  }

struct CallbackOp2
{
    int value;

    __device__ CallbackOp2(int init_value) : value(init_value) {}

    __device__ int operator()(int aggregate)
    {
        int pre_value = value;
        value += aggregate;
        return pre_value;
    }
};

__global__ void ExclusiveScanKernel2(int* data) {
    typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    CallbackOp2 CB(0);
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input = data[threadid];
    int output = 0;

    BlockScan(temp1).ExclusiveScan(input, output, hipcub::Sum(), CB);

    data[threadid] = output;
}
//CHECK:  void ExclusiveScanKernel3(int* data, const sycl::nd_item<3> &item_ct1) {
//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:    int input[4];
//CHECK:    input[0] = data[4 * threadid];
//CHECK:    input[1] = data[4 * threadid + 1];
//CHECK:    input[2] = data[4 * threadid + 2];
//CHECK:    input[3] = data[4 * threadid + 3];
//CHECK:    int output[4];

//CHECK:    dpct::group::exclusive_scan(item_ct1, input, output, 0, sycl::plus<>());

//CHECK:    data[4 * threadid] = output[0];
//CHECK:    data[4 * threadid + 1] = output[1];
//CHECK:    data[4 * threadid + 2] = output[2];
//CHECK:    data[4 * threadid + 3] = output[3];
//CHECK:  }
__global__ void ExclusiveScanKernel3(int* data) {
    typedef hipcub::BlockScan<int, 8, hipcub::BLOCK_SCAN_RAKING, 4, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input[4];
    input[0] = data[4 * threadid];
    input[1] = data[4 * threadid + 1];
    input[2] = data[4 * threadid + 2];
    input[3] = data[4 * threadid + 3];
    int output[4];

    BlockScan(temp1).ExclusiveScan(input, output, 0, hipcub::Sum());

    data[4 * threadid] = output[0];
    data[4 * threadid + 1] = output[1];
    data[4 * threadid + 2] = output[2];
    data[4 * threadid + 3] = output[3];
}

//CHECK:  void InclusiveSumKernel1(int* data, int* aggregate,
//CHECK-NEXT: const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:  int input = data[threadid];
//CHECK:  int output = 0;
//CHECK:  int agg = 0;

//CHECK:  output = dpct::group::inclusive_scan(item_ct1, input, sycl::plus<>(), agg);

//CHECK:  data[threadid] = output;
//CHECK:  aggregate[threadid] = agg;
//CHECK:  }

__global__ void InclusiveSumKernel1(int* data, int* aggregate) {
  typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
  __shared__ typename BlockScan::TempStorage temp1;

    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
  int input = data[threadid];
  int output = 0;
  int agg = 0;

  BlockScan(temp1).InclusiveSum(input, output, agg);

  data[threadid] = output;
  aggregate[threadid] = agg;
}

//CHECK:  struct CallbackOp3
//CHECK:  {
//CHECK:      int value;

//CHECK:      CallbackOp3(int init_value) : value(init_value) {}

//CHECK:      int operator()(int aggregate)
//CHECK:      {
//CHECK:          int pre_value = value;
//CHECK:          value += aggregate;
//CHECK:          return pre_value;
//CHECK:      }
//CHECK:  };

//CHECK:  void InclusiveSumKernel2(int* data, const sycl::nd_item<3> &item_ct1) {
//CHECK:      CallbackOp3 CB(0);
//CHECK:      int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                        + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                        + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                        + item_ct1.get_local_id(2);;
//CHECK:      int input = data[threadid];
//CHECK:      int output = 0;

//CHECK:      output = dpct::group::inclusive_scan(item_ct1, input, sycl::plus<>(), CB);

//CHECK:      data[threadid] = output;
//CHECK:  }

struct CallbackOp3
{
    int value;

    __device__ CallbackOp3(int init_value) : value(init_value) {}

    __device__ int operator()(int aggregate)
    {
        int pre_value = value;
        value += aggregate;
        return pre_value;
    }
};

__global__ void InclusiveSumKernel2(int* data) {
    typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    CallbackOp3 CB(0);
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input = data[threadid];
    int output = 0;

    BlockScan(temp1).InclusiveSum(input, output, CB);

    data[threadid] = output;
}

//CHECK:  void InclusiveSumKernel3(int* data, const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:    int input[4];
//CHECK:    input[0] = data[4 * threadid];
//CHECK:    input[1] = data[4 * threadid + 1];
//CHECK:    input[2] = data[4 * threadid + 2];
//CHECK:    input[3] = data[4 * threadid + 3];
//CHECK:    int output[4];

//CHECK:    dpct::group::inclusive_scan(item_ct1, input, output, sycl::plus<>());

//CHECK:    data[4 * threadid] = output[0];
//CHECK:    data[4 * threadid + 1] = output[1];
//CHECK:    data[4 * threadid + 2] = output[2];
//CHECK:    data[4 * threadid + 3] = output[3];
//CHECK:  }

__global__ void InclusiveSumKernel3(int* data) {
    typedef hipcub::BlockScan<int, 8, hipcub::BLOCK_SCAN_RAKING, 4, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input[4];
    input[0] = data[4 * threadid];
    input[1] = data[4 * threadid + 1];
    input[2] = data[4 * threadid + 2];
    input[3] = data[4 * threadid + 3];
    int output[4];

    BlockScan(temp1).InclusiveSum(input, output);

    data[4 * threadid] = output[0];
    data[4 * threadid + 1] = output[1];
    data[4 * threadid + 2] = output[2];
    data[4 * threadid + 3] = output[3];
}

//CHECK:  void InclusiveScanKernel1(int* data, int* aggregate,
//CHECK-NEXT: const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:  int input = data[threadid];
//CHECK:  int output = 0;
//CHECK:  int agg = 0;

//CHECK:  output = dpct::group::inclusive_scan(item_ct1, input, sycl::plus<>(), agg);

//CHECK:  data[threadid] = output;
//CHECK:  aggregate[threadid] = agg;
//CHECK:  }

__global__ void InclusiveScanKernel1(int* data, int* aggregate) {
  typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
  __shared__ typename BlockScan::TempStorage temp1;

    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
  int input = data[threadid];
  int output = 0;
  int agg = 0;

  BlockScan(temp1).InclusiveScan(input, output, hipcub::Sum(), agg);

  data[threadid] = output;
  aggregate[threadid] = agg;
}

//CHECK:  struct CallbackOp4
//CHECK:  {
//CHECK:      int value;

//CHECK:      CallbackOp4(int init_value) : value(init_value) {}

//CHECK:      int operator()(int aggregate)
//CHECK:      {
//CHECK:          int pre_value = value;
//CHECK:          value += aggregate;
//CHECK:          return pre_value;
//CHECK:      }
//CHECK:  };

//CHECK:  void InclusiveScanKernel2(int* data, const sycl::nd_item<3> &item_ct1) {
//CHECK:      CallbackOp4 CB(0);
//CHECK:      int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                        + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                        + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                        + item_ct1.get_local_id(2);;
//CHECK:      int input = data[threadid];
//CHECK:      int output = 0;

//CHECK:      output = dpct::group::inclusive_scan(item_ct1, input, sycl::plus<>(), CB);

//CHECK:      data[threadid] = output;
//CHECK:  }

struct CallbackOp4
{
    int value;

    __device__ CallbackOp4(int init_value) : value(init_value) {}

    __device__ int operator()(int aggregate)
    {
        int pre_value = value;
        value += aggregate;
        return pre_value;
    }
};

__global__ void InclusiveScanKernel2(int* data) {
    typedef hipcub::BlockScan<int, 16, hipcub::BLOCK_SCAN_RAKING, 8, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    CallbackOp4 CB(0);
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input = data[threadid];
    int output = 0;

    BlockScan(temp1).InclusiveScan(input, output, hipcub::Sum(), CB);

    data[threadid] = output;
}

//CHECK:  void InclusiveScanKernel3(int* data, const sycl::nd_item<3> &item_ct1) {

//CHECK:    int threadid = item_ct1.get_group(2) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1) * item_ct1.get_local_range(0))
//CHECK:                      + item_ct1.get_local_id(0) * (item_ct1.get_local_range(2) * item_ct1.get_local_range(1))
//CHECK:                      + item_ct1.get_local_id(1) * item_ct1.get_local_range(2)
//CHECK:                      + item_ct1.get_local_id(2);;
//CHECK:    int input[4];
//CHECK:    input[0] = data[4 * threadid];
//CHECK:    input[1] = data[4 * threadid + 1];
//CHECK:    input[2] = data[4 * threadid + 2];
//CHECK:    input[3] = data[4 * threadid + 3];
//CHECK:    int output[4];

//CHECK:    dpct::group::inclusive_scan(item_ct1, input, output, sycl::plus<>());

//CHECK:    data[4 * threadid] = output[0];
//CHECK:    data[4 * threadid + 1] = output[1];
//CHECK:    data[4 * threadid + 2] = output[2];
//CHECK:    data[4 * threadid + 3] = output[3];
//CHECK:  }

__global__ void InclusiveScanKernel3(int* data) {
    typedef hipcub::BlockScan<int, 8, hipcub::BLOCK_SCAN_RAKING, 4, 1> BlockScan;
    __shared__ typename BlockScan::TempStorage temp1;
    int threadid = blockIdx.x * (blockDim.x * blockDim.y * blockDim.z)
                      + threadIdx.z * (blockDim.x * blockDim.y)
                      + threadIdx.y * blockDim.x
                      + threadIdx.x;;
    int input[4];
    input[0] = data[4 * threadid];
    input[1] = data[4 * threadid + 1];
    input[2] = data[4 * threadid + 2];
    input[3] = data[4 * threadid + 3];
    int output[4];

    BlockScan(temp1).InclusiveScan(input, output, hipcub::Sum());

    data[4 * threadid] = output[0];
    data[4 * threadid + 1] = output[1];
    data[4 * threadid + 2] = output[2];
    data[4 * threadid + 3] = output[3];
}

int main() {
  bool Result = true;
  int* dev_data = nullptr;
  int* dev_agg = nullptr;

  dim3 GridSize;
  dim3 BlockSize;
  hipMallocManaged(&dev_data, DATA_NUM * sizeof(int));
  hipMallocManaged(&dev_agg, DATA_NUM * sizeof(int));

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect3[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  int agg_expect3[DATA_NUM] = {
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512
  };
  init_data(dev_data, DATA_NUM);
  init_data(dev_agg, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveSumKernel1(dev_data, dev_agg, item_ct1);
  //CHECK:    });
  ExclusiveSumKernel1<<<GridSize, BlockSize>>>(dev_data, dev_agg);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect3, DATA_NUM)) {
    std::cout << "ExclusiveSumKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect3, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }
  if(!verify_data(dev_agg, agg_expect3, DATA_NUM)) {
    std::cout << "ExclusiveSumKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(agg_expect3, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_agg, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect4[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveSumKernel2(dev_data, item_ct1);
  //CHECK:    });
  ExclusiveSumKernel2<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect4, DATA_NUM)) {
    std::cout << "ExclusiveSumKernel2" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect4, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {8, 4, 1};
  int expect5[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveSumKernel3(dev_data, item_ct1);
  //CHECK:    });
  ExclusiveSumKernel3<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect5, DATA_NUM)) {
    std::cout << "ExclusiveSumKernel3" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect5, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }


  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect6[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  int agg_expect6[DATA_NUM] = {
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512
  };
  init_data(dev_data, DATA_NUM);
  init_data(dev_agg, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveScanKernel1(dev_data, dev_agg, item_ct1);
  //CHECK:    });
  ExclusiveScanKernel1<<<GridSize, BlockSize>>>(dev_data, dev_agg);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect6, DATA_NUM)) {
    std::cout << "ExclusiveScanKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect6, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }
  if(!verify_data(dev_agg, agg_expect6, DATA_NUM)) {
    std::cout << "ExclusiveScanKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(agg_expect6, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_agg, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect7[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveScanKernel2(dev_data, item_ct1);
  //CHECK:    });
  ExclusiveScanKernel2<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect7, DATA_NUM)) {
    std::cout << "ExclusiveScanKernel2" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect7, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {8, 4, 1};
  int expect8[DATA_NUM] = {
    0, 0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465,
    496, 528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953,
    2016, 2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465,
    4560, 4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001,
    0, 128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433,
    4592, 4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017,
    10208, 10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625,
    16848, 17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      ExclusiveScanKernel3(dev_data, item_ct1);
  //CHECK:    });
  ExclusiveScanKernel3<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect8, DATA_NUM)) {
    std::cout << "ExclusiveScanKernel3" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect8, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }


  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect9[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  int agg_expect9[DATA_NUM] = {
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512
  };
  init_data(dev_data, DATA_NUM);
  init_data(dev_agg, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveSumKernel1(dev_data, dev_agg, item_ct1);
  //CHECK:    });
  InclusiveSumKernel1<<<GridSize, BlockSize>>>(dev_data, dev_agg);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect9, DATA_NUM)) {
    std::cout << "InclusiveSumKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect9, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }
  if(!verify_data(dev_agg, agg_expect9, DATA_NUM)) {
    std::cout << "InclusiveSumKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(agg_expect9, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_agg, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect10[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveSumKernel2(dev_data, item_ct1);
  //CHECK:    });
  InclusiveSumKernel2<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect10, DATA_NUM)) {
    std::cout << "InclusiveSumKernel2" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect10, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {8, 4, 1};
  int expect11[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveSumKernel3(dev_data, item_ct1);
  //CHECK:    });
  InclusiveSumKernel3<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect11, DATA_NUM)) {
    std::cout << "InclusiveSumKernel3" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect11, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect12[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  int agg_expect12[DATA_NUM] = {
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128, 8128,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512,
    24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512, 24512
  };
  init_data(dev_data, DATA_NUM);
  init_data(dev_agg, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveScanKernel1(dev_data, dev_agg, item_ct1);
  //CHECK:    });
  InclusiveScanKernel1<<<GridSize, BlockSize>>>(dev_data, dev_agg);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect12, DATA_NUM)) {
    std::cout << "InclusiveScanKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect12, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }
  if(!verify_data(dev_agg, agg_expect12, DATA_NUM)) {
    std::cout << "InclusiveScanKernel1" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(agg_expect12, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_agg, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {16, 8, 1};
  int expect13[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveScanKernel2(dev_data, item_ct1);
  //CHECK:    });
  InclusiveScanKernel2<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect13, DATA_NUM)) {
    std::cout << "InclusiveScanKernel2" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect13, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  GridSize = {2};
  BlockSize = {8, 4, 1};
  int expect14[DATA_NUM] = {
    0, 1, 3, 6, 10, 15, 21, 28, 36, 45, 55, 66, 78, 91, 105, 120, 136, 153, 171, 190, 210, 231, 253, 276, 300, 325, 351, 378, 406, 435, 465, 496,
    528, 561, 595, 630, 666, 703, 741, 780, 820, 861, 903, 946, 990, 1035, 1081, 1128, 1176, 1225, 1275, 1326, 1378, 1431, 1485, 1540, 1596, 1653, 1711, 1770, 1830, 1891, 1953, 2016,
    2080, 2145, 2211, 2278, 2346, 2415, 2485, 2556, 2628, 2701, 2775, 2850, 2926, 3003, 3081, 3160, 3240, 3321, 3403, 3486, 3570, 3655, 3741, 3828, 3916, 4005, 4095, 4186, 4278, 4371, 4465, 4560,
    4656, 4753, 4851, 4950, 5050, 5151, 5253, 5356, 5460, 5565, 5671, 5778, 5886, 5995, 6105, 6216, 6328, 6441, 6555, 6670, 6786, 6903, 7021, 7140, 7260, 7381, 7503, 7626, 7750, 7875, 8001, 8128,
    128, 257, 387, 518, 650, 783, 917, 1052, 1188, 1325, 1463, 1602, 1742, 1883, 2025, 2168, 2312, 2457, 2603, 2750, 2898, 3047, 3197, 3348, 3500, 3653, 3807, 3962, 4118, 4275, 4433, 4592,
    4752, 4913, 5075, 5238, 5402, 5567, 5733, 5900, 6068, 6237, 6407, 6578, 6750, 6923, 7097, 7272, 7448, 7625, 7803, 7982, 8162, 8343, 8525, 8708, 8892, 9077, 9263, 9450, 9638, 9827, 10017, 10208,
    10400, 10593, 10787, 10982, 11178, 11375, 11573, 11772, 11972, 12173, 12375, 12578, 12782, 12987, 13193, 13400, 13608, 13817, 14027, 14238, 14450, 14663, 14877, 15092, 15308, 15525, 15743, 15962, 16182, 16403, 16625, 16848,
    17072, 17297, 17523, 17750, 17978, 18207, 18437, 18668, 18900, 19133, 19367, 19602, 19838, 20075, 20313, 20552, 20792, 21033, 21275, 21518, 21762, 22007, 22253, 22500, 22748, 22997, 23247, 23498, 23750, 24003, 24257, 24512
  };
  init_data(dev_data, DATA_NUM);
  //CHECK:  q_ct1.parallel_for(
  //CHECK:    sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
  //CHECK:    [=](sycl::nd_item<3> item_ct1) {
  //CHECK:      InclusiveScanKernel3(dev_data, item_ct1);
  //CHECK:    });
  InclusiveScanKernel3<<<GridSize, BlockSize>>>(dev_data);

  hipDeviceSynchronize();
  if(!verify_data(dev_data, expect14, DATA_NUM)) {
    std::cout << "InclusiveScanKernel3" << " verify failed" << std::endl;
    Result = false;
    std::cout << "expect:" << std::endl;
    print_data(expect14, DATA_NUM);
    std::cout << "current result:" << std::endl;
    print_data(dev_data, DATA_NUM);
  }

  if(Result)
    std::cout << "passed" << std::endl;
  return 0;
}
