#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/blocklevel/blockreduce %S/blockreduce.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/blocklevel/blockreduce/blockreduce.dp.cpp --match-full-lines %s

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define WARP_SIZE 32

void init_data(int* data, int num) {
  for(int i = 0; i < num; i++)
    data[i] = 1;
}
void verify_data(int* data, int num) {
  return;
}
void print_data(int* data, int num) {
  for (int i = 0; i < num; i++) {
    std::cout << data[i] << " ";
  }
  std::cout << std::endl;
}
//CHECK: void SumKernel(int* data, 
//CHECK-NEXT:  sycl::nd_item<3> item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::reduce_over_group(item_ct1.get_group(), input, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void SumKernel(int* data) {
  typedef hipcub::BlockReduce<int, 4> BlockReduce;

  __shared__ typename BlockReduce::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  output = BlockReduce(temp1).Sum(input);
  data[threadid] = output;
}

//CHECK: void ReduceKernel(int* data,
//CHECK-NEXT:  sycl::nd_item<3> item_ct1) {
//CHECK-EMPTY:
//CHECK-NEXT:  int threadid = item_ct1.get_local_id(2);
//CHECK-EMPTY:
//CHECK-NEXT:  int input = data[threadid];
//CHECK-NEXT:  int output = 0;
//CHECK-NEXT:  output = sycl::reduce_over_group(item_ct1.get_group(), input, sycl::plus<>());
//CHECK-NEXT:  data[threadid] = output;
//CHECK-NEXT:}
__global__ void ReduceKernel(int* data) {
  typedef hipcub::BlockReduce<int, 4> BlockReduce;

  __shared__ typename BlockReduce::TempStorage temp1;

  int threadid = threadIdx.x;

  int input = data[threadid];
  int output = 0;
  output = BlockReduce(temp1).Reduce(input, hipcub::Sum());
  data[threadid] = output;
}

int main() {
  int* dev_data = nullptr;

  dim3 GridSize(2);
  dim3 BlockSize(1 , 1, 128);
  int TotalThread = GridSize.x * BlockSize.x * BlockSize.y * BlockSize.z;

  hipMallocManaged(&dev_data, TotalThread * sizeof(int));

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          SumKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  SumKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

  init_data(dev_data, TotalThread);
//CHECK:  q_ct1.parallel_for(
//CHECK-NEXT:        sycl::nd_range<3>(GridSize * BlockSize, BlockSize),
//CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:          ReduceKernel(dev_data, item_ct1);
//CHECK-NEXT:        });
  ReduceKernel<<<GridSize, BlockSize>>>(dev_data);
  hipDeviceSynchronize();
  verify_data(dev_data, TotalThread);

  return 0;
}
