// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_select_flagged %S/device_select_flagged.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_select_flagged/device_select_flagged.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_select_flagged/device_select_flagged.dp.cpp -o %T/devicelevel/device_select_flagged/device_select_flagged.dp.o %}

// CHECK:#include <dpct/dpl_utils.hpp>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <stdio.h>

// CHECK: DPCT1026:{{.*}}
// CHECK: q_ct1.fill(device_select_num, std::distance(device_out, dpct::copy_if(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_flagged, device_out, [](const auto &t) -> bool { return t; })), 1).wait();
void test_1() {
   int n = 5;
  int num = 0;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_flagged = nullptr;
  int *device_select_num = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] =       {1, 2, 3, 4, 5};
  int host_flagged[] =  {0, 1, 0, 1, 0};
  int host_out[5];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMalloc((void **)&device_flagged, n * sizeof(int));
  hipMalloc((void **)&device_select_num, sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipMemcpy(device_flagged, host_flagged, sizeof(host_flagged), hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipMemcpy((void *)&num, (void *)device_select_num, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  hipFree(device_flagged);
  hipFree(device_select_num);
  for (int i = 0; i < num; ++i) {
    printf("%d\n", host_out[i]);
  }
}

// CHECK: DPCT1027:{{.*}}
// CHECK: 0, 0;
// CHECK: q_ct1.fill(device_select_num, std::distance(device_out, dpct::copy_if(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_flagged, device_out, [](const auto &t) -> bool { return t; })), 1).wait();
void test_2() {
  int n = 5;
  int num = 0;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_flagged = nullptr;
  int *device_select_num = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] =       {1, 2, 3, 4, 5};
  int host_flagged[] =  {0, 1, 0, 1, 0};
  int host_out[5];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMalloc((void **)&device_flagged, n * sizeof(int));
  hipMalloc((void **)&device_select_num, sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipMemcpy(device_flagged, host_flagged, sizeof(host_flagged), hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n), 0;
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipMemcpy((void *)&num, (void *)device_select_num, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  hipFree(device_flagged);
  hipFree(device_select_num);
  for (int i = 0; i < num; ++i) {
    printf("%d\n", host_out[i]);
  }
}

// CHECK: dpct::queue_ptr stream = (dpct::queue_ptr)(void *)(uintptr_t)5;
// CHECK: DPCT1026:{{.*}}
// CHECK: stream->fill(device_select_num, std::distance(device_out, dpct::copy_if(oneapi::dpl::execution::device_policy(*stream), device_in, device_in + n, device_flagged, device_out, [](const auto &t) -> bool { return t; })), 1).wait();
void test_3() {
   int n = 5;
  int num = 0;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_flagged = nullptr;
  int *device_select_num = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] =       {1, 2, 3, 4, 5};
  int host_flagged[] =  {0, 1, 0, 1, 0};
  int host_out[5];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMalloc((void **)&device_flagged, n * sizeof(int));
  hipMalloc((void **)&device_select_num, sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipMemcpy(device_flagged, host_flagged, sizeof(host_flagged), hipMemcpyHostToDevice);
  hipStream_t stream = (hipStream_t)(void *)(uintptr_t)5;
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n, stream);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceSelect::Flagged(device_tmp, n_device_tmp, device_in, device_flagged, device_out, device_select_num, n, stream);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipMemcpy((void *)&num, (void *)device_select_num, sizeof(int), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  hipFree(device_flagged);
  hipFree(device_select_num);
  for (int i = 0; i < num; ++i) {
    printf("%d\n", host_out[i]);
  }
}
