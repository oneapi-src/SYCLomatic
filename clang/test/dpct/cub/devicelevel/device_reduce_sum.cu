// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_reduce_sum %S/device_reduce_sum.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_reduce_sum/device_reduce_sum.dp.cpp  %s

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <cassert>

// CHECK: DPCT1026{{.*}}
// CHECK: q_ct1.fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, typename std::iterator_traits<decltype(device_out)>::value_type{}), 1).wait();
void test_1() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}

// CHECK: DPCT1027{{.*}}
// CHECK: q_ct1.fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, typename std::iterator_traits<decltype(device_out)>::value_type{}), 1).wait();
void test_2() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Sum(nullptr, n_device_tmp, device_in, device_out, n), 0;
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}


// CHECK: dpct::queue_ptr stream = (dpct::queue_ptr)(void *)(uintptr_t)5;
// CHECK: DPCT1026{{.*}}
// CHECK: stream->fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(*stream), device_in, device_in + n), 1).wait();
void test_3() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipStream_t stream = (hipStream_t)(void *)(uintptr_t)5;
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n, stream);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n, stream);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}
