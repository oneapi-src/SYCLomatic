// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_reduce_sum %S/device_reduce_sum.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_reduce_sum/device_reduce_sum.dp.cpp --match-full-lines %s

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <cassert>

// CHECK:void test_1() {
// CHECK:dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK:sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK:int n = 10;
// CHECK:int *device_in = nullptr;
// CHECK:int *device_out = nullptr;
// CHECK:int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
// CHECK:int host_out[10];
// CHECK:device_in = sycl::malloc_device<int>(n, q_ct1);
// CHECK:device_out = sycl::malloc_device<int>(n, q_ct1);
// CHECK:q_ct1.memcpy(device_in, (void *)host_in, sizeof(host_in)).wait();
// CHECK:DPCT1026{{.*}}
// CHECK:q_ct1.fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n), 1);
// CHECK:q_ct1.memcpy((void *)host_out, (void *)device_out, sizeof(host_out)).wait();
// CHECK:sycl::free(device_in, q_ct1);
// CHECK:sycl::free(device_out, q_ct1);
// CHECK:assert(host_out[0] == 55);
// CHECK:}
void test_1() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}

// CHECK:void test_2() {
// CHECK:dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK:sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK:int n = 10;
// CHECK:int *device_in = nullptr;
// CHECK:int *device_out = nullptr;
// CHECK:int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
// CHECK:int host_out[10];
// CHECK:device_in = sycl::malloc_device<int>(n, q_ct1);
// CHECK:device_out = sycl::malloc_device<int>(n, q_ct1);
// CHECK:q_ct1.memcpy(device_in, (void *)host_in, sizeof(host_in)).wait();
// CHECK:DPCT1026:{{.*}}
// CHECK:DPCT1026:{{.*}}
// CHECK:DPCT1026:{{.*}}
// CHECK:q_ct1.fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n), 1);
// CHECK:q_ct1.memcpy((void *)host_out, (void *)device_out, sizeof(host_out)).wait();
// CHECK:sycl::free(device_in, q_ct1);
// CHECK:sycl::free(device_out, q_ct1);
// CHECK:assert(host_out[0] == 55);
// CHECK:}
void test_2() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Sum(0, n_device_tmp, device_in, device_out, n);
  hipcub::DeviceReduce::Sum(NULL, n_device_tmp, device_in, device_out, n);
  hipcub::DeviceReduce::Sum(nullptr, n_device_tmp, device_in, device_out, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}

// CHECK:void test_3() {
// CHECK:dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK:sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK:int n = 10;
// CHECK:int *device_in = nullptr;
// CHECK:int *device_out = nullptr;
// CHECK:int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
// CHECK:int host_out[10];
// CHECK:device_in = sycl::malloc_device<int>(n, q_ct1);
// CHECK:device_out = sycl::malloc_device<int>(n, q_ct1);
// CHECK:q_ct1.memcpy(device_in, (void *)host_in, sizeof(host_in)).wait();
// CHECK:DPCT1027{{.*}}
// CHECK:q_ct1.fill(device_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n), 1);
// CHECK:q_ct1.memcpy((void *)host_out, (void *)device_out, sizeof(host_out)).wait();
// CHECK:sycl::free(device_in, q_ct1);
// CHECK:sycl::free(device_out, q_ct1);
// CHECK:assert(host_out[0] == 55);
// CHECK:}
void test_3() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Sum(nullptr, n_device_tmp, device_in, device_out, n), 0;
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceReduce::Sum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
  assert(host_out[0] == 55);
}
