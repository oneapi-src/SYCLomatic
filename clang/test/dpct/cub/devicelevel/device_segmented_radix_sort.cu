// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.2, cuda-11.4
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.2, v11.4
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_segmented_radix_sort %S/device_segmented_radix_sort.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_segmented_radix_sort/device_segmented_radix_sort.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_segmented_radix_sort/device_segmented_radix_sort.dp.cpp -o %T/devicelevel/device_segmented_radix_sort/device_segmented_radix_sort.dp.o %}

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
// CHECK:#include <dpct/dpl_utils.hpp>
#include <hipcub/hipcub.hpp>

int n, num_segments, *d_keys_in, *d_keys_out, *d_values_in, *d_values_out, *d_offsets;

// CHECK:dpct::io_iterator_pair<int *> d_keys(d_keys_in, d_keys_out), d_values(d_values_in, d_values_out);
hipcub::DoubleBuffer<int> d_keys(d_keys_in, d_keys_out), d_values(d_values_in, d_values_out);

void test1(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test1(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, false);
}

void test2(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test2(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, false, 1);
}

void test3(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test3(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, false, 1, 4);
}

void test4(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test4(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(*s), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, false, 1, 4);
}

void test5(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test5(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, true);
}

void test6(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test6(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, true, 1);
}

void test7(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test7(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, true, 1, 4);
}

void test8(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test8(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(*s), d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1, true, 1, 4);
}

void test9(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test9(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, false, true);
}

void test10(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test10(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, false, true, 1);
}

void test11(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test11(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, false, true, 1, 4);
}

void test12(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortPairs(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test12(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(*s), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, false, true, 1, 4);
}

void test13(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test13(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, true, true);
}

void test14(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test14(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, true, true, 1);
}

void test15(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test15(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(q_ct1), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, true, true, 1, 4);
}

void test16(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(nullptr, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortPairsDescending(temp_storage, temp_storage_size, d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test16(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(*s), d_keys, d_values, n, num_segments, d_offsets, d_offsets + 1, true, true, 1, 4);
}

void test17(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test17(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, false);
}

void test18(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test18(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, false, 1);
}

void test19(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test19(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, false, 1, 4);
}

void test20(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test20(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(*s), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, false, 1, 4);
}

void test21(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test21(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, true);
}

void test22(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test22(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, true, 1);
}

void test23(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test23(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, true, 1, 4);
}

void test24(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test24(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_pairs(oneapi::dpl::execution::device_policy(*s), d_keys_in, d_keys_out, n, num_segments, d_offsets, d_offsets + 1, true, 1, 4);
}

void test25(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test25(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, false, true);
}

void test26(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test26(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, false, true, 1);
}

void test27(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test27(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, false, true, 1, 4);
}

void test28(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeys(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeys(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test28(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(*s), d_keys, n, num_segments, d_offsets, d_offsets + 1, false, true, 1, 4);
}

void test29(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1);
// CHECK: void test29(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, true, true);
}

void test30(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1);
// CHECK: void test30(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, true, true, 1);
}

void test31(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4);
// CHECK: void test31(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(q_ct1), d_keys, n, num_segments, d_offsets, d_offsets + 1, true, true, 1, 4);
}

void test32(void) {
  void *temp_storage;
  size_t temp_storage_size;
  hipStream_t s = 0;
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(nullptr, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
  hipMalloc(&temp_storage, temp_storage_size);
  hipcub::DeviceSegmentedRadixSort::SortKeysDescending(temp_storage, temp_storage_size, d_keys, n, num_segments, d_offsets, d_offsets + 1, 1, 4, s);
// CHECK: void test32(void)
// CHECK-NOT: void *temp_storage;
// CHECK-NOT: size_t temp_storage_size;
// CHECK-NOT: hipMalloc(&temp_storage, temp_storage_size);
// CHECK: dpct::queue_ptr s = &q_ct1;
// CHECK: DPCT1026:{{.*}}
// CHECK: dpct::segmented_sort_keys(oneapi::dpl::execution::device_policy(*s), d_keys, n, num_segments, d_offsets, d_offsets + 1, true, true, 1, 4);
}
