// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_inclusive_sum %S/device_inclusive_sum.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_inclusive_sum/device_inclusive_sum.dp.cpp --match-full-lines %s

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>
#include <cassert>

// CHECK: void test_1() {
// CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK: int n = 10;
// CHECK: int *device_in = nullptr;
// CHECK: int *device_out = nullptr;
// CHECK: int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
// CHECK: int host_out[10];
// CHECK: device_in = sycl::malloc_device<int>(n, q_ct1);
// CHECK: device_out = sycl::malloc_device<int>(n, q_ct1);
// CHECK: q_ct1.memcpy(device_in, (void *)host_in, sizeof(host_in)).wait();
// CHECK: DPCT1026:{{.*}}
// CHECK: oneapi::dpl::inclusive_scan(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_out);
// CHECK: q_ct1.memcpy((void *)host_out, (void *)device_out, sizeof(host_out)).wait();
// CHECK: sycl::free(device_in, q_ct1);
// CHECK: sycl::free(device_out, q_ct1);
// CHECK: }
void test_1() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceScan::InclusiveSum(nullptr, n_device_tmp, device_in, device_out, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceScan::InclusiveSum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
}

// void test_2() {
// dpct::device_ext &dev_ct1 = dpct::get_current_device();
// sycl::queue &q_ct1 = dev_ct1.default_queue();
// int n = 10;
// int *device_in = nullptr;
// int *device_out = nullptr;
// int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
// int host_out[10];
// device_in = sycl::malloc_device<int>(n, q_ct1);
// device_out = sycl::malloc_device<int>(n, q_ct1);
// q_ct1.memcpy(device_in, (void *)host_in, sizeof(host_in)).wait();
// DPCT1026:{{.*}}
// DPCT1027:1:{{.*}}
// 0, 0;
// oneapi::dpl::inclusive_scan(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_out);
// q_ct1.memcpy((void *)host_out, (void *)device_out, sizeof(host_out)).wait();
// sycl::free(device_in, q_ct1);
// sycl::free(device_out, q_ct1);
// }
void test_2() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceScan::InclusiveSum(nullptr, n_device_tmp, device_in, device_out, n),0;
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceScan::InclusiveSum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
}
