// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_reduce %S/device_reduce.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_reduce/device_reduce.dp.cpp %s

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>

struct CustomMin {
    template <typename T>
    __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};


// CHECK:void test_1() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: int n = 10;
// CHECK-NEXT: CustomMin op;
// CHECK-NEXT: int *d_in, *d_out;
// CHECK-NEXT: int in[] = {8, 6, 7, 5, -1, 0, 9};
// CHECK-NEXT: d_in = (int *)sycl::malloc_device(sizeof(in), q_ct1);
// CHECK-NEXT: d_out = (int *)sycl::malloc_device(sizeof(in), q_ct1);
// CHECK-NEXT: q_ct1.memcpy((void *)d_in, (void *)in, sizeof(in)).wait();
// CHECK-NEXT: DPCT1026:{{.*}}
// CHECK-NEXT: q_ct1.fill(d_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), d_in, d_in + n, 0, op), 1).wait();
// CHECK-NEXT: q_ct1.memcpy((void *)in, d_out, sizeof(int)).wait();
// CHECK-NEXT: printf("%d\n", in[0]);
// CHECK-NEXT:}
void test_1() {
  int n = 7;
  size_t n_tmp;
  CustomMin op;
  int *d_in, *d_out, *tmp = nullptr;
  int in[] = {8, 6, 7, 5, -1, 0, 9};
  hipMalloc((void **)&d_in, sizeof(in));
  hipMalloc((void **)&d_out, sizeof(in));
  hipMemcpy((void *)d_in, (void *)in, sizeof(in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Reduce(tmp, n_tmp, d_in, d_out, n, op, 0);
  hipMalloc((void **)&tmp, n_tmp);
  hipcub::DeviceReduce::Reduce(tmp, n_tmp, d_in, d_out, n, op, 0);
  hipMemcpy((void *)in, d_out, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", in[0]);
}

// CHECK:void test_2() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: int n = 10;
// CHECK-NEXT: CustomMin op;
// CHECK-NEXT: int *d_in, *d_out;
// CHECK-NEXT: int in[] = {8, 6, 7, 5, -1, 0, 9};
// CHECK-NEXT: d_in = (int *)sycl::malloc_device(sizeof(in), q_ct1);
// CHECK-NEXT: d_out = (int *)sycl::malloc_device(sizeof(in), q_ct1);
// CHECK-NEXT: q_ct1.memcpy((void *)d_in, (void *)in, sizeof(in)).wait();
// CHECK-NEXT: DPCT1026:{{.*}}
// CHECK-NEXT: 0, 0;
// CHECK-NEXT: q_ct1.fill(d_out, oneapi::dpl::reduce(oneapi::dpl::execution::device_policy(q_ct1), d_in, d_in + n, 0, op), 1).wait();
// CHECK-NEXT: q_ct1.memcpy((void *)in, d_out, sizeof(int)).wait();
// CHECK-NEXT: printf("%d\n", in[0]);
// CHECK-NEXT:}
void test_2() {
  int n = 7;
  size_t n_tmp;
  CustomMin op;
  int *d_in, *d_out, *tmp = nullptr;
  int in[] = {8, 6, 7, 5, -1, 0, 9};
  hipMalloc((void **)&d_in, sizeof(in));
  hipMalloc((void **)&d_out, sizeof(in));
  hipMemcpy((void *)d_in, (void *)in, sizeof(in), hipMemcpyHostToDevice);
  hipcub::DeviceReduce::Reduce(tmp, n_tmp, d_in, d_out, n, op, 0);
  hipMalloc((void **)&tmp, n_tmp);
  hipcub::DeviceReduce::Reduce(tmp, n_tmp, d_in, d_out, n, op, 0);
  hipMemcpy((void *)in, d_out, sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", in[0]);
}
