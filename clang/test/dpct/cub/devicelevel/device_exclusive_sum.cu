// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_exclusive_sum %S/device_exclusive_sum.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_exclusive_sum/device_exclusive_sum.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_exclusive_sum/device_exclusive_sum.dp.cpp -o %T/devicelevel/device_exclusive_sum/device_exclusive_sum.dp.o %}

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  hipError_t err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

// CHECK:DPCT1026:{{.*}}
// CHECK:oneapi::dpl::exclusive_scan(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_out, typename std::iterator_traits<decltype(device_in)>::value_type{});
void test_1() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceScan::ExclusiveSum(device_tmp, n_device_tmp, device_in, device_out, n);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceScan::ExclusiveSum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
}

// CHECK: DPCT1027:{{.*}}
// CHECK: DPCT_CHECK_ERROR(0), 0;
// CHECK: oneapi::dpl::exclusive_scan(oneapi::dpl::execution::device_policy(q_ct1), device_in, device_in + n, device_out, typename std::iterator_traits<decltype(device_in)>::value_type{});
void test_2() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipcub::DeviceScan::ExclusiveSum(device_tmp, n_device_tmp, device_in, device_out, n), 0;
  hipMalloc((void **)&device_tmp, n_device_tmp);
  hipcub::DeviceScan::ExclusiveSum((void *)device_tmp, n_device_tmp, device_in, device_out, n);
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
}

// CHECK: dpct::queue_ptr stream = (dpct::queue_ptr)(void *)(uintptr_t)5;
// CHECK: DPCT1026:{{.*}}
// CHECK: cudaCheck(DPCT_CHECK_ERROR(oneapi::dpl::exclusive_scan(oneapi::dpl::execution::device_policy(*stream), device_in, device_in + n, device_out, typename std::iterator_traits<decltype(device_in)>::value_type{})));
void test_3() {
  int n = 10;
  int *device_in = nullptr;
  int *device_out = nullptr;
  int *device_tmp = nullptr;
  size_t n_device_tmp = 0;
  int host_in[] = {1, 2, 3, 4, 5, 6, 7, 8, 9, 10};
  int host_out[10];
  hipMalloc((void **)&device_in, n * sizeof(int));
  hipMalloc((void **)&device_out, n * sizeof(int));
  hipMemcpy(device_in, (void *)host_in, sizeof(host_in), hipMemcpyHostToDevice);
  hipStream_t stream = (hipStream_t)(void *)(uintptr_t)5;
  hipcub::DeviceScan::ExclusiveSum(device_tmp, n_device_tmp, device_in, device_out, n, stream);
  hipMalloc((void **)&device_tmp, n_device_tmp);
  cudaCheck(hipcub::DeviceScan::ExclusiveSum((void *)device_tmp, n_device_tmp, device_in, device_out, n, stream));
  hipMemcpy((void *)host_out, (void *)device_out, sizeof(host_out), hipMemcpyDeviceToHost);
  hipFree(device_in);
  hipFree(device_out);
  hipFree(device_tmp);
}
