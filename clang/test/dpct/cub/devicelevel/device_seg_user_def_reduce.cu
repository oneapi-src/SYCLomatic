// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_seg_user_def_reduce %S/device_seg_user_def_reduce.cu --use-experimental-features=user-defined-reductions --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.cpp %s --check-prefixes=CHECK,CHECK-DPCT1092
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.cpp -o %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.o %}
// RUN: FileCheck --input-file %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.cpp %s --check-prefixes=CHECK,CHECK-DPCT1026
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.cpp -o %T/devicelevel/device_seg_user_def_reduce/device_seg_user_def_reduce.dp.o %}
// RUN: rm -rf %T/devicelevel/device_seg_user_def_reduce/

#include <iostream>
#include <vector>

#include <hip/hip_runtime.h>
#include <hipcub/hipcub.hpp>

#define DATA_NUM 100

template<typename T = int>
void init_data(T* data, int num) {
  for(int i = 0; i < num; i++)
    data[i] = i;
}

template<typename T = int>
bool verify_data(T* data, T* expect, int num, int step = 1) {
  for(int i = 0; i < num; i = i + step) {
    if(data[i] != expect[i]) {
      return false;
    }
  }
  return true;
}

template<typename T = int>
void print_data(T* data, int num) {
  for (int i = 0; i < num; i++) {
    std::cout << data[i] << ", ";
    if((i+1)%32 == 0)
        std::cout << std::endl;
  }
  std::cout << std::endl;
}

//CHECK:  struct UserMin
//CHECK:  {
//CHECK:    template <typename T>
//CHECK:    __dpct_inline__
//CHECK:    T operator()(const T &a, const T &b) const {
//CHECK:        return (b < a) ? b : a;
//CHECK:    }
//CHECK:  };

struct UserMin
{
  template <typename T>
  __device__ __host__ __forceinline__
  T operator()(const T &a, const T &b) const {
      return (b < a) ? b : a;
  }
};

//CHECK:  bool test_reduce_1(){
//CHECK:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
//CHECK:  sycl::queue &q_ct1 = dev_ct1.in_order_queue();
//CHECK:    int          num_segments = 10;
//CHECK:    int          *device_offsets;
//CHECK:    int          *device_in;
//CHECK:    int          *device_out;
//CHECK:    UserMin      min_op;
//CHECK:    int          initial_value = INT_MAX;
//CHECK:    int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};
//CHECK:    device_offsets = sycl::malloc_shared<int>((num_segments + 1), q_ct1);
//CHECK:    device_in = sycl::malloc_shared<int>(DATA_NUM, q_ct1);
//CHECK:    device_out = sycl::malloc_shared<int>(num_segments, q_ct1);
//CHECK:    init_data(device_in, DATA_NUM);
//CHECK:    for(int i = 0; i < num_segments + 1; i++) {
//CHECK:      device_offsets[i] = i * 10;
//CHECK:    }
//CHECK-DPCT1092:    DPCT1092:{{[0-9]+}}: Consider replacing work-group size 128 with different value for specific hardware for better performance.
//CHECK-DPCT1026:    DPCT1026:{{[0-9]+}}: The call to hipcub::DeviceSegmentedReduce::Reduce was removed because this call is redundant in SYCL.
//CHECK:    dpct::device::experimental::segmented_reduce<128>(q_ct1, device_in, device_out, num_segments, device_offsets, device_offsets + 1, min_op, initial_value);
//CHECK:    dev_ct1.queues_wait_and_throw();
//CHECK:    if(!verify_data(device_out, expect, num_segments)) {
//CHECK:      std::cout << "Reduce" << " verify failed" << std::endl;
//CHECK:      std::cout << "expect:" << std::endl;
//CHECK:      print_data<int>(expect, num_segments);
//CHECK:      std::cout << "current result:" << std::endl;
//CHECK:      print_data<int>(device_out, num_segments);
//CHECK:      return false;
//CHECK:    }
//CHECK:    return true;
//CHECK:  }
bool test_reduce_1(){
  int          num_segments = 10;
  int          *device_offsets;
  int          *device_in;
  int          *device_out;
  UserMin      min_op;
  int          initial_value = INT_MAX;
  void     *temp_storage = NULL;
  size_t   temp_storage_size = 0;
  int expect[DATA_NUM] = {0, 10, 20, 30, 40, 50, 60, 70, 80, 90};

  hipMallocManaged(&device_offsets, (num_segments + 1) * sizeof(int));
  hipMallocManaged(&device_in, DATA_NUM * sizeof(int));
  hipMallocManaged(&device_out, num_segments * sizeof(int));
  init_data(device_in, DATA_NUM);
  for(int i = 0; i < num_segments + 1; i++) {
    device_offsets[i] = i * 10;
  }
  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, min_op, initial_value);

  hipMalloc(&temp_storage, temp_storage_size);

  hipcub::DeviceSegmentedReduce::Reduce(temp_storage, temp_storage_size, device_in, device_out, num_segments, device_offsets, device_offsets + 1, min_op, initial_value);

  hipDeviceSynchronize();

  if(!verify_data(device_out, expect, num_segments)) {
    std::cout << "Reduce" << " verify failed" << std::endl;
    std::cout << "expect:" << std::endl;
    print_data<int>(expect, num_segments);
    std::cout << "current result:" << std::endl;
    print_data<int>(device_out, num_segments);
    return false;
  }
  return true;
}

int main() {
  if(test_reduce_1()) {
    std::cout << "Pass" << std::endl;
  }
  return 0;
}
