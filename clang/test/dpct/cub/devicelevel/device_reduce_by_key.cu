// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_reduce_by_key %S/device_reduce_by_key.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_reduce_by_key/device_reduce_by_key.dp.cpp %s

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <iostream>

struct CustomMin {
    template <typename T>
    __device__ __forceinline__
    T operator()(const T &a, const T &b) const {
        return (b < a) ? b : a;
    }
};


// CHECK:void test_1() {
// CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT:  int n = 8;
// CHECK-NEXT:  CustomMin op;
// CHECK-NEXT:  int unq[n], agg[n], num;
// CHECK-NEXT:  int key[] = {0, 2, 2, 9, 5, 5, 5, 8};
// CHECK-NEXT:  int val[] = {0, 7, 1, 6, 2, 5, 3, 4};
// CHECK-NEXT:  int *d_key, *d_val, *d_unq, *d_agg, *d_num;
// CHECK-NEXT:  d_key = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_val = (int *)sycl::malloc_device(sizeof(val), q_ct1);
// CHECK-NEXT:  d_unq = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_agg = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_num = sycl::malloc_device<int>(1, q_ct1);
// CHECK-NEXT:  q_ct1.memcpy(d_key, key, sizeof(key)){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(d_val, val, sizeof(val)){{.*}};
// CHECK-NEXT:  DPCT1026{{.*}}
// CHECK-NEXT:  q_ct1.fill(d_num, std::distance(d_unq, oneapi::dpl::reduce_by_key(oneapi::dpl::execution::device_policy(q_ct1), d_key, d_key + n, d_val, d_unq, d_agg, std::equal_to<typename std::iterator_traits<decltype(d_key)>::value_type>(), op).first), 1).wait();
// CHECK-NEXT:  q_ct1.memcpy(&num, d_num, sizeof(int)){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(unq, d_unq, sizeof(int) * num){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(agg, d_agg, sizeof(int) * num){{.*}};
// CHECK-NEXT:}
void test_1() {
  int n = 8;
  CustomMin op;
  int unq[n], agg[n], num;
  int key[] = {0, 2, 2, 9, 5, 5, 5, 8};
  int val[] = {0, 7, 1, 6, 2, 5, 3, 4};
  int *d_key, *d_val, *d_unq, *d_agg, *d_num;
  hipMalloc(&d_key, sizeof(key));
  hipMalloc(&d_val, sizeof(val));
  hipMalloc(&d_unq, sizeof(key));
  hipMalloc(&d_agg, sizeof(key));
  hipMalloc(&d_num, sizeof(int));
  hipMemcpy(d_key, key, sizeof(key), hipMemcpyHostToDevice);
  hipMemcpy(d_val, val, sizeof(val), hipMemcpyHostToDevice);
  void *tmp = nullptr;
  size_t n_tmp = 0;
  hipcub::DeviceReduce::ReduceByKey(tmp, n_tmp, d_key, d_unq, d_val, d_agg, d_num, op, n);
  hipMalloc(&tmp, n_tmp);
  hipcub::DeviceReduce::ReduceByKey(tmp, n_tmp, d_key, d_unq, d_val, d_agg, d_num, op, n);
  hipMemcpy(&num, d_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(unq, d_unq, sizeof(int) * num, hipMemcpyDeviceToHost);
  hipMemcpy(agg, d_agg, sizeof(int) * num, hipMemcpyDeviceToHost);
}

// CHECK:void test_2() {
// CHECK-NEXT:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT:  int n = 8;
// CHECK-NEXT:  CustomMin op;
// CHECK-NEXT:  int unq[n], agg[n], num;
// CHECK-NEXT:  int key[] = {0, 2, 2, 9, 5, 5, 5, 8};
// CHECK-NEXT:  int val[] = {0, 7, 1, 6, 2, 5, 3, 4};
// CHECK-NEXT:  int *d_key, *d_val, *d_unq, *d_agg, *d_num;
// CHECK-NEXT:  d_key = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_val = (int *)sycl::malloc_device(sizeof(val), q_ct1);
// CHECK-NEXT:  d_unq = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_agg = (int *)sycl::malloc_device(sizeof(key), q_ct1);
// CHECK-NEXT:  d_num = sycl::malloc_device<int>(1, q_ct1);
// CHECK-NEXT:  q_ct1.memcpy(d_key, key, sizeof(key)){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(d_val, val, sizeof(val)){{.*}};
// CHECK-NEXT:  DPCT1026{{.*}}
// CHECK-NEXT:  0, 0;
// CHECK-NEXT:  q_ct1.fill(d_num, std::distance(d_unq, oneapi::dpl::reduce_by_key(oneapi::dpl::execution::device_policy(q_ct1), d_key, d_key + n, d_val, d_unq, d_agg, std::equal_to<typename std::iterator_traits<decltype(d_key)>::value_type>(), op).first), 1).wait();
// CHECK-NEXT:  q_ct1.memcpy(&num, d_num, sizeof(int)){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(unq, d_unq, sizeof(int) * num){{.*}};
// CHECK-NEXT:  q_ct1.memcpy(agg, d_agg, sizeof(int) * num){{.*}};
// CHECK-NEXT:}
void test_2() {
  int n = 8;
  CustomMin op;
  int unq[n], agg[n], num;
  int key[] = {0, 2, 2, 9, 5, 5, 5, 8};
  int val[] = {0, 7, 1, 6, 2, 5, 3, 4};
  int *d_key, *d_val, *d_unq, *d_agg, *d_num;
  hipMalloc(&d_key, sizeof(key));
  hipMalloc(&d_val, sizeof(val));
  hipMalloc(&d_unq, sizeof(key));
  hipMalloc(&d_agg, sizeof(key));
  hipMalloc(&d_num, sizeof(int));
  hipMemcpy(d_key, key, sizeof(key), hipMemcpyHostToDevice);
  hipMemcpy(d_val, val, sizeof(val), hipMemcpyHostToDevice);
  void *tmp = nullptr;
  size_t n_tmp = 0;
  hipcub::DeviceReduce::ReduceByKey(tmp, n_tmp, d_key, d_unq, d_val, d_agg, d_num, op, n), 0;
  hipMalloc(&tmp, n_tmp);
  hipcub::DeviceReduce::ReduceByKey(tmp, n_tmp, d_key, d_unq, d_val, d_agg, d_num, op, n);
  hipMemcpy(&num, d_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy(unq, d_unq, sizeof(int) * num, hipMemcpyDeviceToHost);
  hipMemcpy(agg, d_agg, sizeof(int) * num, hipMemcpyDeviceToHost);
}
