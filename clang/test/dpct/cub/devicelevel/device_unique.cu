// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -in-root %S -out-root %T/devicelevel/device_unique %S/device_unique.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/devicelevel/device_unique/device_unique.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/devicelevel/device_unique/device_unique.dp.cpp -o %T/devicelevel/device_unique/device_unique.dp.o %}

// Missing wait() synchronization for memcpy with dependencies

// CHECK:#include <oneapi/dpl/execution>
// CHECK:#include <oneapi/dpl/algorithm>
#include <hipcub/hipcub.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>

#define N 8

// CHECK: DPCT1026{{.*}}
// CHECK: q_ct1.fill(d_selected_num, std::distance(d_out, oneapi::dpl::unique_copy(oneapi::dpl::execution::device_policy(q_ct1), d_in, d_in + N, d_out)), 1).wait();
void test_1() {
  int h_in[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int h_out[N] = {0};
  int *d_in = nullptr;
  int *d_out = nullptr;
  int *d_temp = nullptr;
  int *d_selected_num = nullptr;
  int h_selected_num = 0;
  size_t d_temp_size = 0;
  hipMalloc((void **)&d_in, sizeof(h_in));
  hipMalloc((void **)&d_out, sizeof(h_out));
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)h_in, sizeof(h_in), hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Unique(nullptr, d_temp_size, d_in, d_out, d_selected_num, N);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceSelect::Unique((void *)d_temp, d_temp_size, d_in, d_out, d_selected_num, N);
  hipMemcpy((void *)&h_selected_num, (void *)d_selected_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)h_out, (void *)d_out, h_selected_num * sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", h_selected_num);
  for (int i = 0; i < h_selected_num; ++i)
    printf("%d\n", h_out[i]);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp);
  hipFree(d_selected_num);
}

// CHECK: DPCT1027:{{.*}}
// CHECK: 0, 0;
// CHECK: q_ct1.fill(d_selected_num, std::distance(d_out, oneapi::dpl::unique_copy(oneapi::dpl::execution::device_policy(q_ct1), d_in, d_in + N, d_out)), 1).wait();
void test_2() {
  int h_in[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int h_out[N] = {0};
  int *d_in = nullptr;
  int *d_out = nullptr;
  int *d_temp = nullptr;
  int *d_selected_num = nullptr;
  int h_selected_num = 0;
  size_t d_temp_size = 0;
  hipMalloc((void **)&d_in, sizeof(h_in));
  hipMalloc((void **)&d_out, sizeof(h_out));
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)h_in, sizeof(h_in), hipMemcpyHostToDevice);
  hipcub::DeviceSelect::Unique(nullptr, d_temp_size, d_in, d_out, d_selected_num, N), 0;
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceSelect::Unique((void *)d_temp, d_temp_size, d_in, d_out, d_selected_num, N);
  hipMemcpy((void *)&h_selected_num, (void *)d_selected_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)h_out, (void *)d_out, h_selected_num * sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", h_selected_num);
  for (int i = 0; i < h_selected_num; ++i)
    printf("%d\n", h_out[i]);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp);
  hipFree(d_selected_num);
}

// CHECK: dpct::queue_ptr stream = (dpct::queue_ptr)(void *)(uintptr_t)5;
// CHECK: DPCT1026{{.*}}
// CHECK: stream->fill(d_selected_num, std::distance(d_out, oneapi::dpl::unique_copy(oneapi::dpl::execution::device_policy(*stream), d_in, d_in + N, d_out)), 1).wait();
void test_3() {
  int h_in[N] = {0, 2, 2, 9, 5, 5, 5, 8};
  int h_out[N] = {0};
  int *d_in = nullptr;
  int *d_out = nullptr;
  int *d_temp = nullptr;
  int *d_selected_num = nullptr;
  int h_selected_num = 0;
  size_t d_temp_size = 0;
  hipMalloc((void **)&d_in, sizeof(h_in));
  hipMalloc((void **)&d_out, sizeof(h_out));
  hipMalloc((void **)&d_selected_num, sizeof(int));
  hipMemcpy((void *)d_in, (void *)h_in, sizeof(h_in), hipMemcpyHostToDevice);
  hipStream_t stream = (hipStream_t)(void *)(uintptr_t)5;
  hipcub::DeviceSelect::Unique(nullptr, d_temp_size, d_in, d_out, d_selected_num, N, stream);
  hipMalloc((void **)&d_temp, d_temp_size);
  hipcub::DeviceSelect::Unique((void *)d_temp, d_temp_size, d_in, d_out, d_selected_num, N, stream);
  hipMemcpy((void *)&h_selected_num, (void *)d_selected_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)h_out, (void *)d_out, h_selected_num * sizeof(int), hipMemcpyDeviceToHost);
  printf("%d\n", h_selected_num);
  for (int i = 0; i < h_selected_num; ++i)
    printf("%d\n", h_out[i]);
  hipFree(d_in);
  hipFree(d_out);
  hipFree(d_temp);
  hipFree(d_selected_num);
}
