#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/intrinsic/row_major_tid %S/row_major_tid.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/intrinsic/row_major_tid/row_major_tid.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/intrinsic/row_major_tid/row_major_tid.dp.cpp -o %T/intrinsic/row_major_tid/row_major_tid.dp.o %}

// CHECK:#include <sycl/sycl.hpp>
// CHECK:#include <dpct/dpct.hpp>
#include <hipcub/hipcub.hpp>

__global__ void kernel(int *res) {
  // CHECK: *res = item_ct1.get_local_linear_id();
  *res = hipcub::RowMajorTid(blockDim.x, blockDim.y, blockDim.z);
}
