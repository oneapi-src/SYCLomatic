#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/intrinsic/bfe_bfi %S/bfe_bfi.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/intrinsic/bfe_bfi/bfe_bfi.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/intrinsic/bfe_bfi/bfe_bfi.dp.cpp -o %T/intrinsic/bfe_bfi/bfe_bfi.dp.o %}

// CHECK:#include <sycl/sycl.hpp>
// CHECK:#include <dpct/dpct.hpp>
#include <hipcub/hipcub.hpp>
#include <cstdint>

__global__ void kernel() {
  int32_t i32 = 0;
  uint32_t u32 = 0;
  int64_t i64 = 0;
  uint64_t u64 = 0;
  uint32_t bit_start = 1;
  uint32_t num_bits = 4;
  uint32_t res;

  // CHECK: dpct::bfe_safe(i32, bit_start, num_bits);
  // CHECK-NEXT: dpct::bfe_safe(u32, bit_start, num_bits);
  // CHECK-NEXT: dpct::bfe_safe(i64, bit_start, num_bits);
  // CHECK-NEXT: dpct::bfe_safe(u64, bit_start, num_bits);
  // CHECK-NEXT: res = dpct::bfi_safe<unsigned>(res, u32, u32, bit_start);
  hipcub::BFE(i32, bit_start, num_bits);
  hipcub::BFE(u32, bit_start, num_bits);
  hipcub::BFE(i64, bit_start, num_bits);
  hipcub::BFE(u64, bit_start, num_bits);
  hipcub::BFI(res, u32, u32, bit_start, num_bits);
}
