#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/intrinsic/sync_stream %S/sync_stream.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/intrinsic/sync_stream/sync_stream.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/intrinsic/sync_stream/sync_stream.dp.cpp -o %T/intrinsic/sync_stream/sync_stream.dp.o %}

// CHECK:#include <sycl/sycl.hpp>
// CHECK:#include <dpct/dpct.hpp>
#include <hipcub/hipcub.hpp>
#include <limits>
#include <stdio.h>

__global__ void kernel(int *res) {
  // CHECK: q_ct1.wait();
  hipcub::SyncStream(0);
}

int main() {
  hipStream_t s;
  hipStreamCreate(&s);

  // CHECK: s->wait();
  hipcub::SyncStream(s);

  // CHECK: q_ct1.wait();
  hipcub::SyncStream(0);

  // CHECK: q_ct1.wait();
  hipcub::SyncStream((hipStream_t)(uintptr_t)1);

  // CHECK: q_ct1.wait();
  hipcub::SyncStream((hipStream_t)(uintptr_t)2);

  hipStreamDestroy(s);
  return 0;
}
