#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/intrinsic/load %S/load.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/intrinsic/load/load.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/intrinsic/load/load.dp.cpp -o %T/intrinsic/load/load.dp.o %}

// CHECK:#include <sycl/sycl.hpp>
// CHECK:#include <dpct/dpct.hpp>
// CHECK:#include <dpct/group_utils.hpp>
#include <hipcub/hipcub.hpp>

__global__ void TestLoadStriped(int *d_data) {
  int thread_data[4];
  // CHECK: dpct::group::load_direct_striped(item_ct1, d_data, thread_data);
  hipcub::LoadDirectStriped<128>(threadIdx.x, d_data, thread_data);
}

__global__ void BlockedToStripedKernel(int *d_data) {
  int thread_data[4];
  // CHECK: dpct::group::load_direct_blocked(item_ct1, d_data, thread_data);
  hipcub::LoadDirectBlocked(threadIdx.x, d_data, thread_data);
}
