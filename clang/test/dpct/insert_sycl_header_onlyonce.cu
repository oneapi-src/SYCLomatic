// RUN: dpct --format-range=none -out-root %T/insert_sycl_header_onlyonce %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/insert_sycl_header_onlyonce/insert_sycl_header_onlyonce.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/insert_sycl_header_onlyonce/insert_sycl_header_onlyonce.dp.cpp -o %T/insert_sycl_header_onlyonce/insert_sycl_header_onlyonce.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK: #include <stdio.h>
// CHECK-NOT:#include <sycl/sycl.hpp>
#include <stdio.h>
#include <hip/hip_runtime.h>

int main(){
  return 0;
}


