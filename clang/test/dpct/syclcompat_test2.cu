// RUN: dpct --use-syclcompat --format-range=none --out-root %T/syclcompat_test2 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/syclcompat_test2/syclcompat_test2.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -DBUILD_TEST -c -fsycl %T/syclcompat_test2/syclcompat_test2.dp.cpp -o %T/syclcompat_test2/syclcompat_test2.dp.o %}


#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>

void f1_1() {
  hipfftHandle plan1;
  size_t* work_size;
  int odist;
  int ostride;
  int * onembed;
  int idist;
  int istride;
  int* inembed;
  int * n;
  // CHECK: plan1->commit(&syclcompat::get_current_device().default_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

void f1_2() {
  float *f;
  // CHECK: f = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc(&f, sizeof(float));
  hipfftHandle plan1;
  size_t* work_size;
  int odist;
  int ostride;
  int * onembed;
  int idist;
  int istride;
  int* inembed;
  int * n;
  // CHECK: plan1->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

void f2() {
  hiprandRngType_t rngT;
  hiprandGenerator_t rng;
  // CHECK: rng = dpct::rng::create_host_rng(rngT, *syclcompat::cpu_device().default_queue());
  hiprandCreateGeneratorHost(&rng, rngT);
}

__constant__ float const_float[10][10];

// CHECK: void k3(syclcompat::accessor<float, syclcompat::memory_region::constant, 2> const_float) {
__global__ void k3() {
  float ff = const_float[1][1];
  double d;
  sincospi(d, &d, &d);
}

void f3() {
  k3<<<1, 1>>>();
}

void f4() {
  int atomicSupported;
  int dev_id = 0;
  // CHECK: atomicSupported = syclcompat::get_device(dev_id).is_native_host_atomic_supported();
  hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);
}

void f5() {
  float *f;
// CHECK: #if DPCT_COMPAT_RT_VERSION
// CHECK-NEXT:   f = (float *)sycl::malloc_device(4, syclcompat::get_default_queue());
// CHECK-NEXT: #endif
#if CUDART_VERSION
  hipMalloc(&f, 4);
#endif
}
