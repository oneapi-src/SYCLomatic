// RUN: dpct --use-syclcompat --format-range=none --out-root %T/syclcompat_test2 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/syclcompat_test2/syclcompat_test2.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -DNO_BUILD_TEST -c -fsycl %T/syclcompat_test2/syclcompat_test2.dp.cpp -o %T/syclcompat_test2/syclcompat_test2.dp.o %}


#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>
#include <hiprand/hiprand.h>
#include <hipsolver.h>

void f1_1() {
  hipfftHandle plan1;
  size_t* work_size;
  int odist;
  int ostride;
  int * onembed;
  int idist;
  int istride;
  int* inembed;
  int * n;
  // CHECK: plan1->commit(syclcompat::get_current_device().default_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

void f1_2() {
  float *f;
  // CHECK: f = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc(&f, sizeof(float));
  hipfftHandle plan1;
  size_t* work_size;
  int odist;
  int ostride;
  int * onembed;
  int idist;
  int istride;
  int* inembed;
  int * n;
  // CHECK: plan1->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

void f2() {
  hiprandRngType_t rngT;
  hiprandGenerator_t rng;
  // CHECK: rng = dpct::rng::create_host_rng(rngT, *syclcompat::cpu_device().default_queue());
  hiprandCreateGeneratorHost(&rng, rngT);
}

__constant__ float const_float[10][10];

// CHECK: void k3(syclcompat::accessor<float, syclcompat::memory_region::constant, 2> const_float) {
__global__ void k3() {
  float ff = const_float[1][1];
  double d;
  // CHECK: d = sycl::sincos(d * (3.141592653589793115998), sycl::address_space_cast<sycl::access::address_space::generic_space, sycl::access::decorated::yes>(&d));
  sincospi(d, &d, &d);
}

void f3() {
  k3<<<1, 1>>>();
}

void f4() {
  int atomicSupported;
  int dev_id = 0;
  // CHECK: atomicSupported = syclcompat::get_device(dev_id).is_native_host_atomic_supported();
  hipDeviceGetAttribute(&atomicSupported, hipDeviceAttributeHostNativeAtomicSupported, dev_id);
}

void f5() {
  float *f;
// CHECK: #if DPCT_COMPAT_RT_VERSION
// CHECK-NEXT:   f = (float *)sycl::malloc_device(4, syclcompat::get_default_queue());
// CHECK-NEXT: #endif
#if CUDART_VERSION
  hipMalloc(&f, 4);
#endif
}

void f6_1() {
  hipsolverHandle_t handle;
  // CHECK: handle = syclcompat::get_current_device().default_queue();
  hipsolverDnCreate(&handle);
}

void f6_2() {
  float *f;
  // CHECK: f = sycl::malloc_device<float>(1, q_ct1);
  hipMalloc(&f, sizeof(float));
  hipsolverHandle_t handle;
  // CHECK: handle = &q_ct1;
  hipsolverDnCreate(&handle);
}

void f7() {
  hipEvent_t e;
  // CHECK: /*
  // CHECK-NEXT: DPCT1131:{{[0-9]+}}: The migration of "cudaEventQuery" is not currently supported with SYCLcompat. Please adjust the code manually.
  // CHECK-NEXT: */
#ifndef NO_BUILD_TEST
  hipEventQuery(e);
#endif
}

void f8() {
  // CHECK: syclcompat::queue_ptr s = &q_ct1;
  hipStream_t s = hipStreamLegacy;
  // CHECK: syclcompat::queue_ptr s1 = &q_ct1;
  hipStream_t s1 = hipStreamDefault;
}

void f8_1() {
  // CHECK: syclcompat::queue_ptr s = syclcompat::get_current_device().default_queue();
  hipStream_t s = hipStreamLegacy;
}
