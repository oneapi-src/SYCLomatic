#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/thrust-for-h2o4gpu %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only -fno-delayed-template-parsing -ferror-limit=50
// RUN: FileCheck --input-file %T/thrust-for-h2o4gpu/thrust-for-h2o4gpu.dp.cpp --match-full-lines %s


// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT: #include <oneapi/dpl/algorithm>
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <dpct/dpl_utils.hpp>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/reduce.h>
#include <algorithm>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/device_ptr.h>
// for cuda 12.0
#include <thrust/count.h>
#include <thrust/iterator/constant_iterator.h>
#include <thrust/sort.h>
#include <thrust/unique.h>

template <typename T> struct is_even {
  __host__ __device__ bool operator()(T x) {
    return (static_cast<unsigned int>(x) & 1) == 0;
  }
};

template <typename T> struct absolute_value {
  __host__ __device__ void operator()(T &x) const { x = (x > 0 ? x : -x); }
};

//CHECK: template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
//CHECK-NEXT: void copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, Predicate pred, Iterator3 result2)
//CHECK-NEXT: {
//CHECK-NEXT:   *result2 = std::copy_if(exec, first, last, result1, pred);
//CHECK-NEXT: }
template<typename ExecutionPolicy, typename Iterator1, typename Iterator2, typename Predicate, typename Iterator3>
__global__ void copy_if_kernel(ExecutionPolicy exec, Iterator1 first, Iterator1 last, Iterator2 result1, Predicate pred, Iterator3 result2)
{
  *result2 = thrust::copy_if(exec, first, last, result1, pred);
}

template<typename ExecutionPolicy>
void copy_if_device(ExecutionPolicy exec)
{
  size_t n = 1000;

  //CHECK: std::vector<int>   h_data (n, 1);
  //CHECK-NEXT: dpct::device_vector<int> d_data = h_data;
  //CHECK-NEXT: dpct::device_vector<int> d_result(n);
  //CHECK-NEXT: dpct::device_vector<typename dpct::device_vector<int>::iterator> dd(1);
  thrust::host_vector<int>   h_data (n, 1);
  thrust::device_vector<int> d_data = h_data;
  thrust::device_vector<int> d_result(n);
  thrust::device_vector<typename thrust::device_vector<int>::iterator> dd(1);

  //CHECK: dpct::get_in_order_queue().submit(
  //CHECK-NEXT:  [&](sycl::handler &cgh) {
  //CHECK-NEXT:    auto d_data_begin_ct1 = d_data.begin();
  //CHECK-NEXT:    auto d_data_end_ct2 = d_data.end();
  //CHECK-NEXT:    auto d_result_begin_ct3 = d_result.begin();
  //CHECK-NEXT:    auto dd_begin_ct5 = dd.begin();
  //CHECK-EMPTY:
  //CHECK-NEXT:    cgh.parallel_for(
  //CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  //CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:        copy_if_kernel(exec, d_data_begin_ct1, d_data_end_ct2, d_result_begin_ct3, is_even<int>(), dd_begin_ct5);
  //CHECK-NEXT:      });
  //CHECK-NEXT:  });
  copy_if_kernel<<<1,1>>>(exec, d_data.begin(), d_data.end(), d_result.begin(), is_even<int>(), dd.begin());
}

template<typename T>
struct isfoo_test {
    __host__ __device__ bool operator()(const T a) const { return true; }
};

//CHECK: struct is_even_2 {
//CHECK-NEXT:   bool operator()(int x)  const {
//CHECK-NEXT:     return (static_cast<unsigned int>(x) & 1) == 0;
//CHECK-NEXT:   }
//CHECK-NEXT: };
//CHECK-NEXT: struct is_even_3 {
//CHECK-NEXT:   bool operator()(int x)  const;
//CHECK-NEXT: };
//CHECK-NEXT: bool is_even_3::operator()(int x)  const {
//CHECK-NEXT:   return (static_cast<unsigned int>(x) & 1) == 0;
//CHECK-NEXT: }
struct is_even_2 {
  __host__ __device__ bool operator()(int x) {
    return (static_cast<unsigned int>(x) & 1) == 0;
  }
};
struct is_even_3 {
  __host__ __device__ bool operator()(int x);
};
__host__ __device__ bool is_even_3::operator()(int x) {
  return (static_cast<unsigned int>(x) & 1) == 0;
}
template<class T>
struct is_even_4 {
  __host__ __device__ bool operator()(T x) {
    return (static_cast<unsigned int>(x) & 1) == 0;
  }
};

struct my_math
{
//CHECK: int operator()(const int &r) const{ return r+1;}
__host__ __device__ int operator()(const int &r) const{ return r+1;}
};

void foo() {
  //CHECK: copy_if_device(oneapi::dpl::execution::seq);
  copy_if_device(thrust::seq);

  //CHECK: std::vector<int> h_data(10, 1);
  //CHECK-NEXT: std::vector<int> h_result(10);
  //CHECK-NEXT: dpct::device_vector<int> *data[10];
  //CHECK-NEXT: dpct::device_vector<int> d_new_potential_centroids(10);
  //CHECK-NEXT: auto range = dpct::make_counting_iterator(0);
  thrust::host_vector<int> h_data(10, 1);
  thrust::host_vector<int> h_result(10);
  thrust::device_vector<int> *data[10];
  thrust::device_vector<int> d_new_potential_centroids(10);
  auto range = thrust::make_counting_iterator(0);
  thrust::counting_iterator<int> last = range + 10;
  //CHECK: std::copy_if(oneapi::dpl::execution::seq, h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());
  //CHECK-NEXT: std::copy_if(oneapi::dpl::execution::seq, h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());
  //CHECK-NEXT: dpct::copy_if(oneapi::dpl::execution::make_device_policy(q_ct1), (*data[0]).begin(), (*data[0]).end(), range, d_new_potential_centroids.begin(), [=] (int idx) { return true; });
  //CHECK-NEXT: dpct::copy_if(oneapi::dpl::execution::seq, range, last, (*data[0]).begin(), (*data[0]).end(), oneapi::dpl::identity());
  thrust::copy_if(h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());
  thrust::copy_if(thrust::seq, h_data.begin(), h_data.end(), h_result.begin(), is_even<int>());
  thrust::copy_if((*data[0]).begin(), (*data[0]).end(), range, d_new_potential_centroids.begin(),[=] __device__(int idx) { return true; });
  thrust::copy_if(range, last, (*data[0]).begin(), (*data[0]).end(), thrust::identity<int>());

  //CHECK: std::vector<dpct::device_vector<int>> d(10);
  //CHECK-NEXT: auto t = dpct::make_counting_iterator(0);
  //CHECK-NEXT: auto min_costs_ptr = dpct::get_raw_pointer(d[0].data());
  //CHECK-NEXT: int pot_cent_num = std::count_if(oneapi::dpl::execution::seq, t, t + 10, [=] (int idx) { return true;});
  std::vector<thrust::device_vector<int>> d(10);
  auto t = thrust::make_counting_iterator(0);
  auto min_costs_ptr = thrust::raw_pointer_cast(d[0].data());
  int pot_cent_num = thrust::count_if(t, t + 10, [=] __device__(int idx) { return true;});

  {
  float *_de = NULL;
  float fill_value = 0.0;

  //CHECK: dpct::device_pointer<float> dev_ptr = dpct::get_device_pointer(static_cast<float *>(&_de[0]));
  //CHECK-NEXT: std::fill(oneapi::dpl::execution::make_device_policy(q_ct1), dev_ptr, dev_ptr + 10, fill_value);
  //CHECK-NEXT: std::fill_n(oneapi::dpl::execution::make_device_policy(q_ct1), dev_ptr, 10, fill_value);
  //CHECK-NEXT: float M_inner = dpct::inner_product(oneapi::dpl::execution::make_device_policy(q_ct1), dev_ptr, dev_ptr + 10, dev_ptr, 0.0f);
  thrust::device_ptr<float> dev_ptr = thrust::device_pointer_cast(static_cast<float *>(&_de[0]));
  thrust::fill(dev_ptr, dev_ptr + 10, fill_value);
  thrust::fill_n(dev_ptr, 10, fill_value);
  float M_inner = thrust::inner_product(dev_ptr, dev_ptr + 10, dev_ptr, 0.0f);
  }

 {
  //CHECK: dpct::device_vector<double> t;
  //CHECK-NEXT: std::for_each(oneapi::dpl::execution::make_device_policy(q_ct1), t.begin(), t.end(), absolute_value<double>());
  thrust::device_vector<double> t;
  thrust::for_each( t.begin(), t.end(), absolute_value<double>());
 }

 {
  //CHECK: dpct::device_vector<int> t;
  //CHECK-NEXT: std::for_each(oneapi::dpl::execution::make_device_policy(q_ct1), t.begin(), t.end(), is_even_2());
  //CHECK-NEXT: std::for_each(oneapi::dpl::execution::make_device_policy(q_ct1), t.begin(), t.end(), is_even_3());
  //CHECK-NEXT: std::for_each(oneapi::dpl::execution::make_device_policy(q_ct1), t.begin(), t.end(), is_even_4<int>());
  thrust::device_vector<int> t;
  thrust::for_each(t.begin(), t.end(), is_even_2());
  thrust::for_each(t.begin(), t.end(), is_even_3());
  thrust::for_each(t.begin(), t.end(), is_even_4<int>());
 }

 {
  //CHECK: int min = std::min(1, 2);
  //CHECK-NEXT: int max = std::max(1, 2);
  int min = thrust::min(1, 2);
  int max = thrust::max(1, 2);
 }

 {
  //CHECK: dpct::device_vector<int> a, b, c;
  //CHECK-NEXT: dpct::sort(oneapi::dpl::execution::make_device_policy(q_ct1), a.begin(), b.end(), c.begin());
  thrust::device_vector<int> a, b, c;
  thrust::sort_by_key(a.begin(), b.end(), c.begin());
 }

 {
  const int N = 1000;
  //CHECK: dpct::device_vector<float> t1(N);
  //CHECK-NEXT: dpct::device_vector<float> t2(N);
  //CHECK-NEXT: dpct::device_vector<float> t3(N);
  //CHECK-NEXT: std::transform(oneapi::dpl::execution::make_device_policy(q_ct1), t1.begin(), t1.end(), t2.begin(), t3.begin(), std::divides<float>());
  //CHECK-NEXT: std::transform(oneapi::dpl::execution::make_device_policy(q_ct1), t1.begin(), t1.end(), t2.begin(), t3.begin(), std::multiplies<float>());
  //CHECK-NEXT: std::transform(oneapi::dpl::execution::make_device_policy(q_ct1), t1.begin(), t1.end(), t2.begin(), t3.begin(), std::plus<float>());
  thrust::device_vector<float> t1(N);
  thrust::device_vector<float> t2(N);
  thrust::device_vector<float> t3(N);
  thrust::transform(t1.begin(), t1.end(), t2.begin(), t3.begin(), thrust::divides<float>());
  thrust::transform(t1.begin(), t1.end(), t2.begin(), t3.begin(), thrust::multiplies<float>());
  thrust::transform(t1.begin(), t1.end(), t2.begin(), t3.begin(), thrust::plus<float>());
 }

 {
    //CHECK: dpct::device_vector<int> data(4);
    //CHECK-NEXT: std::transform(oneapi::dpl::execution::make_device_policy(q_ct1), data.begin(), data.end(), dpct::make_constant_iterator(10), data.begin(), std::divides<int>());
    thrust::device_vector<int> data(4);
    thrust::transform(data.begin(), data.end(), thrust::make_constant_iterator(10), data.begin(), thrust::divides<int>());
 }

 {
    //CHECK: std::tuple<int, const char *> t(13, "foo");
    //CHECK-NEXT: std::cout << "The 1st value of t is " << std::get<0>(t) << std::endl;
    //CHECK-NEXT: auto ret = std::make_tuple(3, 4);
    thrust::tuple<int, const char *> t(13, "foo");
    std::cout << "The 1st value of t is " << thrust::get<0>(t) << std::endl;
    auto ret = thrust::make_tuple(3, 4);
 }

 {
  //CHECK: dpct::device_vector<int> int_in(3);
  //CHECK-NEXT: dpct::device_vector<float> float_in(3);
  //CHECK-NEXT: typedef dpct::device_vector<int>::iterator int_iterator;
  //CHECK-NEXT: typedef dpct::device_vector<float>::iterator float_iterator;
  //CHECK-NEXT: typedef std::tuple<int_iterator, float_iterator> iterator_tuple;
  //CHECK-NEXT: dpct::zip_iterator<iterator_tuple> ret = oneapi::dpl::make_zip_iterator(std::make_tuple(int_in.begin(), float_in.begin()));
  //CHECK-NEXT: auto arg = std::make_tuple(int_in.begin(), float_in.begin());
  //CHECK-NEXT: dpct::zip_iterator<iterator_tuple> ret_1 = oneapi::dpl::make_zip_iterator(arg);
  thrust::device_vector<int> int_in(3);
  thrust::device_vector<float> float_in(3);
  typedef thrust::device_vector<int>::iterator int_iterator;
  typedef thrust::device_vector<float>::iterator float_iterator;
  typedef thrust::tuple<int_iterator, float_iterator> iterator_tuple;
  thrust::zip_iterator<iterator_tuple> ret = thrust::make_zip_iterator(thrust::make_tuple(int_in.begin(), float_in.begin()));
  auto arg = thrust::make_tuple(int_in.begin(), float_in.begin());
  thrust::zip_iterator<iterator_tuple> ret_1 = thrust::make_zip_iterator(arg);
 }

 {
   // CHECK: int a;
   // CHECK-NEXT: double b;
   // CHECK-NEXT: std::tie(a, b) = std::make_tuple(1, 2.0);
   int a;
   double b;
   thrust::tie(a, b) = thrust::make_tuple(1, 2.0);
 }

 {
   // CHECK: using TupleTy = std::tuple<int, const char *>;
   using TupleTy = thrust::tuple<int, const char *>;
   // CHECK: using EleType_0 = typename std::tuple_element<0, TupleTy>::type;
   using EleType_0 = typename thrust::tuple_element<0, TupleTy>::type;
   // CHECK: using EleType_1 = std::tuple_element<1, std::tuple<int, const char *>>::type;
   using EleType_1 = thrust::tuple_element<1, thrust::tuple<int, const char *>>::type;
   // CHECK: typedef typename std::tuple_element<0, std::tuple<int, typename std::tuple_element<1, std::tuple<int, const char *>>::type>>::type EleType_2;
   typedef typename thrust::tuple_element<0, thrust::tuple<int, typename thrust::tuple_element<1, thrust::tuple<int, const char *>>::type>>::type EleType_2;
   static_assert(std::is_same<int, EleType_0>::value, "EleType_0 should be alias of int");
   static_assert(std::is_same<const char *, EleType_1>::value, "EleType_1 should be alias of const char *");
   static_assert(std::is_same<int, EleType_2>::value, "EleType_2 should be alias of int");

   // CHECK: typename std::tuple_element<0, TupleTy>::type v0;
   typename thrust::tuple_element<0, TupleTy>::type v0;
   // CHECK: extern std::tuple_element<0, TupleTy>::type bar1();
   extern thrust::tuple_element<0, TupleTy>::type bar1();
   // CHECK: extern void foo1(typename std::tuple_element<0, TupleTy>::type v1);
   extern void foo1(typename thrust::tuple_element<0, TupleTy>::type v1);

   struct {
     // CHECK: std::tuple_element<0, std::tuple<int, const char *>>::type m = 10;
     thrust::tuple_element<0, thrust::tuple<int, const char *>>::type m = 10;
     // CHECK: std::tuple_element<1, std::tuple<int, const char *>>::type s = "struct st";
     thrust::tuple_element<1, thrust::tuple<int, const char *>>::type s = "struct st";
   } st;
   std::cout << st.m << ", " << st.s << std::endl;
 }

 {
   // CHECK: using TupleTy = std::tuple<int, double, const char *>;
   using TupleTy = thrust::tuple<int, double, const char *>;
   // CHECK: const int size = std::tuple_size<TupleTy>::value;
   const int size = thrust::tuple_size<TupleTy>::value;
   static_assert(size == 3, "TupleTy size shoud be 3");
 }

 {
  //CHECK: int x =  137;
  //CHECK-NEXT: int y = -137;
  //CHECK-NEXT: oneapi::dpl::maximum<int> mx;
  //CHECK-NEXT: int value = mx(x,y);
  int x =  137;
  int y = -137;
  thrust::maximum<int> mx;
  int value = mx(x,y);
 }

 {
  int data[10];
  //CHECK: dpct::device_pointer<int> begin = dpct::get_device_pointer(&data[0]);
  //CHECK-NEXT: dpct::device_pointer<int> end=begin + 10;
  //CHECK-NEXT: bool h_result = std::transform_reduce(oneapi::dpl::execution::make_device_policy(q_ct1), begin, end, 0, std::plus<bool>(), isfoo_test<int>());
  //CHECK-NEXT: bool h_result_1 = std::transform_reduce(oneapi::dpl::execution::seq, begin, end, 0, std::plus<bool>(), isfoo_test<int>());
  //CHECK-NEXT: auto ptrs = std::make_tuple(begin, end);
  //CHECK-NEXT: int num = std::get<1>(ptrs) - std::get<0>(ptrs);
  thrust::device_ptr<int> begin = thrust::device_pointer_cast(&data[0]);
  thrust::device_ptr<int> end=begin + 10;
  bool h_result = thrust::transform_reduce(begin, end, isfoo_test<int>(), 0, thrust::plus<bool>());
  bool h_result_1 = thrust::transform_reduce(thrust::seq, begin, end, isfoo_test<int>(), 0, thrust::plus<bool>());
  auto ptrs = thrust::make_tuple(begin, end);
  int num = thrust::get<1>(ptrs) - thrust::get<0>(ptrs);
 }

{
  int *dev_a = NULL, *dev_b = NULL;
  hipStream_t stream;
  my_math c;
  //CHECK: std::transform(oneapi::dpl::execution::make_device_policy(*stream), dev_a, dev_a + 10, dev_b, c);
  thrust::transform(thrust::cuda::par.on(stream),dev_a,dev_a + 10,dev_b,c);
}

{
  int values[10] = {1, 0, 1, 0, 1, 0, 1, 0, 1, 0};
  thrust::device_vector<int> d_values(values, values + 10);
  int map[10] = {0, 2, 4, 6, 8, 1, 3, 5, 7, 9};
  thrust::device_vector<int> d_map(map, map + 10);
  thrust::device_vector<int> d_output(10);
  // CHECK: dpct::gather(oneapi::dpl::execution::make_device_policy(q_ct1), d_map.begin(), d_map.end(), d_values.begin(), d_output.begin());
  // CHECK-NEXT: dpct::gather(oneapi::dpl::execution::make_device_policy(q_ct1), d_map.begin(), d_map.end(), d_values.begin(), d_output.begin());
  thrust::gather(d_map.begin(), d_map.end(), d_values.begin(), d_output.begin());
  thrust::gather(thrust::device, d_map.begin(), d_map.end(), d_values.begin(),d_output.begin());
}

{
  int values[10] = {1, 0, 1, 0, 1, 0, 1, 0, 1, 0};
  thrust::host_vector<int> h_values(values, values + 10);
  int map[10] = {0, 2, 4, 6, 8, 1, 3, 5, 7, 9};
  thrust::host_vector<int> h_map(map, map + 10);
  thrust::host_vector<int> h_output(10);

  // CHECK: dpct::gather(oneapi::dpl::execution::seq, h_map.begin(), h_map.end(), h_values.begin(), h_output.begin());
  // CHECK-NEXT: dpct::gather(oneapi::dpl::execution::seq, h_map.begin(), h_map.end(), h_values.begin(), h_output.begin());
  thrust::gather(thrust::seq, h_map.begin(), h_map.end(), h_values.begin(),h_output.begin());
  thrust::gather(h_map.begin(), h_map.end(), h_values.begin(),h_output.begin());
}

{
  int values[10] = {1, 0, 1, 0, 1, 0, 1, 0, 1, 0};
  thrust::device_vector<int> d_values(values, values + 10);
  int map[10] = {0, 5, 1, 6, 2, 7, 3, 8, 4, 9};
  thrust::device_vector<int> d_map(map, map + 10);
  thrust::device_vector<int> d_output(10);

  // CHECK: dpct::scatter(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());
  // CHECK-NEXT: dpct::scatter(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());
  thrust::scatter(d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());
  thrust::scatter(thrust::device, d_values.begin(), d_values.end(), d_map.begin(), d_output.begin());
}

{
  int values[10] = {1, 0, 1, 0, 1, 0, 1, 0, 1, 0};
  thrust::host_vector<int> h_values(values, values + 10);
  int map[10] = {0, 5, 1, 6, 2, 7, 3, 8, 4, 9};
  thrust::host_vector<int> h_map(map, map + 10);
  thrust::host_vector<int> h_output(10);

  // CHECK: dpct::scatter(oneapi::dpl::execution::seq, h_values.begin(), h_values.end(), h_map.begin(), h_output.begin());
  // CHECK-NEXT: dpct::scatter(oneapi::dpl::execution::seq, h_values.begin(), h_values.end(), h_map.begin(), h_output.begin());
  thrust::scatter(thrust::seq, h_values.begin(), h_values.end(), h_map.begin(), h_output.begin());
  thrust::scatter(h_values.begin(), h_values.end(), h_map.begin(), h_output.begin());
}

{
  const int N = 7;
  int A[N] = {1, 3, 3, 3, 2, 2, 1}; // input keys
  int B[N] = {9, 8, 7, 6, 5, 4, 3}; // input values

  thrust::device_vector<int> d_keys(A, A + N);
  thrust::device_vector<int> d_values(B, B + N);
  thrust::device_vector<int> d_output_keys(N);
  thrust::device_vector<int> d_output_values(N);
  thrust::equal_to<int> binary_pred;

  typedef thrust::pair<thrust::device_vector<int>::iterator,
                       thrust::device_vector<int>::iterator>
      iter_pair;
  thrust::device_vector<iter_pair> new_last_vec(1);
  iter_pair new_last;

  thrust::pair<int *, int *> new_end;

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin(), binary_pred);
  *new_last_vec.begin() = thrust::unique_by_key_copy(thrust::device, d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin(), binary_pred);

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin(), binary_pred);
  *new_last_vec.begin() = thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin(), binary_pred);

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin());
  *new_last_vec.begin() = thrust::unique_by_key_copy(d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin());

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin());
  *new_last_vec.begin() = thrust::unique_by_key_copy(thrust::device, d_keys.begin(), d_keys.end(), d_values.begin(), d_output_keys.begin(), d_output_values.begin());
}

{
  const int N = 7;
  int A[N] = {1, 3, 3, 3, 2, 2, 1}; // input keys
  int B[N] = {9, 8, 7, 6, 5, 4, 3}; // input values

  thrust::host_vector<int> h_keys(A, A + N);
  thrust::host_vector<int> h_values(B, B + N);
  thrust::host_vector<int> h_output_keys(N);
  thrust::host_vector<int> h_output_values(N);
  thrust::equal_to<int> binary_pred;

  typedef thrust::pair<thrust::host_vector<int>::iterator,
                       thrust::host_vector<int>::iterator>
      iter_pair;
  thrust::host_vector<iter_pair> new_last_vec(1);
  iter_pair new_last;

  thrust::pair<int *, int *> new_end;

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin(), binary_pred);
  *new_last_vec.begin() = thrust::unique_by_key_copy(thrust::seq, h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin(), binary_pred);

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin(), binary_pred);
  *new_last_vec.begin() = thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin(), binary_pred);

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin());
  *new_last_vec.begin() = thrust::unique_by_key_copy(h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin());

  // CHECK: *new_last_vec.begin() = dpct::unique_copy(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), h_output_keys.begin(), h_output_values.begin());
  *new_last_vec.begin() = thrust::unique_by_key_copy(thrust::seq, h_keys.begin(), h_keys.end(), h_values.begin(),h_output_keys.begin(), h_output_values.begin());
}
}

// CHECK: const std::vector<float> transform(
// CHECK-NEXT:     const std::vector<int>& src, size_t width, size_t height, size_t pitch)
// CHECK-NEXT: {
// CHECK-NEXT:     const std::vector<float> result(100, 0);
// CHECK-NEXT:     return result;
// CHECK-NEXT: }
const thrust::host_vector<float> transform(
    const thrust::host_vector<int>& src, size_t width, size_t height, size_t pitch)
{
    const thrust::host_vector<float> result(100, 0);
    return result;
}

// CHECK: template <typename T>
// CHECK-NEXT: const std::vector<float> transformT(
// CHECK-NEXT:     const std::vector<T>& src, size_t width, size_t height, size_t pitch)
// CHECK-NEXT: {
// CHECK-NEXT:     const std::vector<float> result(100, 0);
// CHECK-NEXT:     return result;
// CHECK-NEXT: }
template <typename T>
const thrust::host_vector<float> transformT(
    const thrust::host_vector<T>& src, size_t width, size_t height, size_t pitch)
{
    const thrust::host_vector<float> result(100, 0);
    return result;
}

// CHECK: const dpct::device_vector<float> transform(
// CHECK-NEXT:     const dpct::device_vector<int>& src, size_t width, size_t height, size_t pitch)
// CHECK-NEXT: {
// CHECK-NEXT:     const dpct::device_vector<float> result(100, 0);
// CHECK-NEXT:     return result;
// CHECK-NEXT: }
const thrust::device_vector<float> transform(
    const thrust::device_vector<int>& src, size_t width, size_t height, size_t pitch)
{
    const thrust::device_vector<float> result(100, 0);
    return result;
}

// CHECK: template <typename T>
// CHECK-NEXT: const dpct::device_vector<float> transformT(
// CHECK-NEXT:     const dpct::device_vector<T>& src, size_t width, size_t height, size_t pitch)
// CHECK-NEXT: {
// CHECK-NEXT:     const dpct::device_vector<float> result(100, 0);
// CHECK-NEXT:     return result;
// CHECK-NEXT: }
template <typename T>
const thrust::device_vector<float> transformT(
    const thrust::device_vector<T>& src, size_t width, size_t height, size_t pitch)
{
    const thrust::device_vector<float> result(100, 0);
    return result;
}

void test(){
    // CHECK: const std::vector<float> d_actual;
    const thrust::host_vector<float> d_actual;
    // CHECK: const dpct::device_vector<float> d_actual2;
    const thrust::device_vector<float> d_actual2;
}

struct make_pair_functor
{
  template<typename T1, typename T2>
  __host__ __device__ thrust::pair<T1,T2> operator()(const T1 &x, const T2 &y)
  {
    return thrust::make_pair(x,y);
  } // end operator()()
}; // end make_pair_functor

// CHECK: typedef std::pair<int,int> P;
// CHECK-NEXT: std::pair<int,int> P1;
typedef thrust::pair<int,int> P;
thrust::pair<int,int> P1;

class AbstractInput {
public:
  AbstractInput() {}
  ~AbstractInput() {}

  template <size_t index> int *&getOutputNode() {
   // CHECK:    return *std::get<index>(m_pOutputNodes);
    return *std::get<index>(m_pOutputNodes);
  }

private:
  int *m_pOutputNodes;
};

__global__ void kernel1(){
  int a[10];

  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), a, a + 9);
  thrust::sort(thrust::device, a, a + 9);
}
template<typename Itr>
void mysort(Itr Beg, Itr End){
  hipStream_t s1;
  thrust::host_vector<int> h_vec(10);
  thrust::device_vector<int> d_vec(10);

  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::seq, Beg, End);
  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::seq, Beg, End);
  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), Beg, End);
  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(*s1), Beg, End);
  thrust::sort(Beg, End);
  thrust::sort(thrust::host, Beg, End);
  thrust::sort(thrust::device, Beg, End);
  thrust::sort(thrust::cuda::par.on(s1), Beg, End);

  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::seq, h_vec.begin(), h_vec.end());
  // CHECK:  oneapi::dpl::sort(oneapi::dpl::execution::seq, h_vec.begin(), h_vec.end());
  thrust::sort(thrust::host, h_vec.begin(), h_vec.end());
  thrust::sort(h_vec.begin(), h_vec.end());

  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(*s1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec.begin(), d_vec.end());
  thrust::sort(thrust::cuda::par.on(s1), d_vec.begin(), d_vec.end());
  thrust::sort(thrust::device, d_vec.begin(), d_vec.end());
  thrust::sort(d_vec.begin(), d_vec.end());
}


typedef hipStream_t FooType;
template <typename T> class Container {

public:
  Container(FooType stream) { m_Stream = stream; };
  FooType getStream() const { return m_Stream; }

  FooType m_Stream;
};

template <typename InputType, typename OutputType>
void myfunction(const std::shared_ptr<const Container<InputType>> &inImageData,
                int *dev_a, int *dev_b) {
  // CHECK: std::transform(oneapi::dpl::execution::make_device_policy(*inImageData->getStream()), dev_a, dev_a + 10, dev_b, my_math());
  thrust::transform(thrust::cuda::par.on(inImageData->getStream()), dev_a, dev_a + 10, dev_b, my_math());
}

template <typename InputType, typename OutputType>
void myfunction2(FooType stream, int *dev_a, int *dev_b) {
  // CHECK: std::transform(oneapi::dpl::execution::make_device_policy(*stream), dev_a, dev_a + 10, dev_b, my_math());
  thrust::transform(thrust::cuda::par.on(stream), dev_a, dev_a + 10, dev_b, my_math());
}

int main(void){
  thrust::host_vector<int> h_vec(10);
  thrust::device_vector<int> d_vec(10);
  hipStream_t s1;

  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::seq, h_vec.begin(), h_vec.end());
  // CHECK:   oneapi::dpl::sort(oneapi::dpl::execution::seq, h_vec.begin(), h_vec.end());
  thrust::sort(thrust::host, h_vec.begin(), h_vec.end());
  thrust::sort(h_vec.begin(), h_vec.end());

  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(*s1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec.begin(), d_vec.end());
  // CHECK: oneapi::dpl::sort(oneapi::dpl::execution::seq, d_vec.begin(), d_vec.end());
  thrust::sort(thrust::cuda::par.on(s1), d_vec.begin(), d_vec.end());
  thrust::sort(thrust::device, d_vec.begin(), d_vec.end());
  thrust::sort(d_vec.begin(), d_vec.end());
  thrust::sort(thrust::cuda::par, d_vec.begin(), d_vec.end());
  thrust::sort(thrust::host, d_vec.begin(), d_vec.end());

  int x = 1;
  int y = 2;
  // CHECK: std::swap(x, y);
  thrust::swap(x, y);
  // CHECK: auto c = std::make_pair(1, 2);
  auto c = thrust::make_pair(1, 2);

  return 0;
}

template <bool is_ture> void foo() {
  hipStream_t stream;
  int index_count = 1;

  // CHECK:  std::for_each(oneapi::dpl::execution::make_device_policy(*stream), oneapi::dpl::counting_iterator<size_t>(0), oneapi::dpl::counting_iterator<size_t>(index_count), [=] (size_t i) {
  // CHECK-NEXT:      if /*constexpr*/ (is_true) {
  // CHECK-NEXT:        i = 1;
  // CHECK-NEXT:      }
  // CHECK-NEXT:    });
  thrust::for_each(
      thrust::cuda::par.on(stream), thrust::counting_iterator<size_t>(0),
      thrust::counting_iterator<size_t>(index_count), [=] __device__(size_t i) {
        if /*constexpr*/ (is_true) {
          i = 1;
        }
      });
}
