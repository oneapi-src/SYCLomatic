// RUN: dpct --enable-profiling --format-range=none --usm-level=none -out-root %T/cuda-event-api-enable-profiling %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-event-api-enable-profiling/cuda-event-api-enable-profiling.dp.cpp --match-full-lines %s

// CHECK:#define DPCT_PROFILING_ENABLED
// CHECK-NEXT: #define DPCT_USM_LEVEL_NONE
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <stdio.h>

#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

//CHECK: #define CudaEvent(X) X = new sycl::event()
#define CudaEvent(X) hipEventCreate(&X)

#define MY_CHECKER(CALL) do {                           \
  hipError_t Error = CALL;                             \
  if (Error != hipSuccess) {                           \
  }                                                     \
} while(0)

__global__ void kernelFunc()
{
}

int main(int argc, char* argv[]) {
// CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK: dpct::event_ptr start, stop;
// CHECK-EMPTY:
// CHECK: start = new sycl::event();
// CHECK: stop = new sycl::event();
// CHECK-EMPTY:
// CHECK-NEXT: float elapsed_time;
// CHECK-EMPTY:
// CHECK-NEXT: dev_ct1.queues_wait_and_throw();
// CHECK-EMPTY:
// CHECK-NEXT: int blocks = 32, threads = 32;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK: printf("<<<\n");
// CHECK-NEXT: CudaEvent(start);
// CHECK-NEXT: printf(">>>\n");
  printf("<<<\n");
  CudaEvent(start);
  printf(">>>\n");


// CHECK: MY_ERROR_CHECKER((start = new sycl::event(), 0));
// CHECK: int et = (stop = new sycl::event(), 0);
  MY_ERROR_CHECKER(hipEventCreate(&start));
  hipError_t et = hipEventCreate(&stop);


  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK:   *start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(start, 0);

  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();


// CHECK: *start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(start, 0);

// CHECK: MY_ERROR_CHECKER((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_ERROR_CHECKER(hipEventRecord(start, 0));

// CHECK: if (0)
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:   */
// CHECK-NEXT:   MY_ERROR_CHECKER((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

  // kernel call with sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK:   *stop = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(stop, 0);

// CHECK: MY_ERROR_CHECKER((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));


// CHECK:   MY_ERROR_CHECKER((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  if (1)
    MY_ERROR_CHECKER(hipEventRecord(stop, 0));

  // kernel call without sync
// CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:           kernelFunc();
// CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

// CHECK: *stop = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(stop, 0);

// CHECK: /*
// CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT: */
// CHECK-NEXT: MY_ERROR_CHECKER((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));

// CHECK: if (0)
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:   */
// CHECK-NEXT:   MY_ERROR_CHECKER((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

// CHECK:  MY_ERROR_CHECKER((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_ERROR_CHECKER(hipEventRecord(start));

  // kernel call without sync
// CHECK:  DPCT1049:{{[0-9a-f]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:  */
// CHECK-NEXT:  q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          kernelFunc();
// CHECK-NEXT:        });
  kernelFunc<<<blocks,threads>>>();

// CHECK:  *stop = q_ct1.ext_oneapi_submit_barrier();
// CHECK-NEXT:  stop->wait_and_throw();
// CHECK-NEXT:  elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);

// CHECK: /*
// CHECK-NEXT: DPCT1003:{{[0-9a-z]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: MY_ERROR_CHECKER((elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f, 0));
  MY_ERROR_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

}

void foo() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK:  MY_CHECKER((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_CHECKER(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
// CHECK: MY_CHECKER((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  MY_CHECKER(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);

// CHECK: MY_CHECKER((elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f, 0));
  MY_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);

  {
  // CHECK: int err = (*start = q_ct1.ext_oneapi_submit_barrier(), 0);
    hipError_t err = hipEventRecord(start, 0);
  // CHECK: err = (*stop = q_ct1.ext_oneapi_submit_barrier(), 0);
    err = hipEventRecord(stop, 0);
    if (hipSuccess != err) {
      printf("%s\n", hipGetErrorString( err));
    }
  }
}

void fun(int) {}

void bar() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

// CHECK: fun((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  fun(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
// CHECK: fun((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  fun(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);
// CHECK: fun((elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f, 0));
  fun(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

struct Node {
 // CHECK: dpct::event_ptr start;
 hipEvent_t start;
 // CHECK: dpct::event_ptr end;
 hipEvent_t end;
 // CHECK: dpct::event_ptr *ev[100];
 hipEvent_t *ev[100];
 // CHECK: dpct::event_ptr events[100];
 hipEvent_t events[100];
 // CHECK: dpct::event_ptr *p_events;
 hipEvent_t *p_events;
};

void foo2(Node *n) {
  float elapsed_time;

// CHECK: *n->start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->start, 0);
// CHECK: *n->start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->start, 0);
  // do something
// CHECK: *n->end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->end, 0);
// CHECK: *n->end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(n->end, 0);
// CHECK: elapsed_time = (n->end->get_profiling_info<sycl::info::event_profiling::command_end>() - n->start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventElapsedTime(&elapsed_time, n->start, n->end);
  {
    int errorCode;
  // CHECK: MY_CHECKER((*n->start = q_ct1.ext_oneapi_submit_barrier(), 0));
    MY_CHECKER(hipEventRecord(n->start, 0));
  // CHECK: errorCode = (*n->start = q_ct1.ext_oneapi_submit_barrier(), 0);
    errorCode = hipEventRecord(n->start, 0);
  }

  Node node;
// CHECK: *node.start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.start, 0);
// CHECK: *node.start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.start, 0);
  // do something
// CHECK: *node.end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.end, 0);
// CHECK: *node.end = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(node.end, 0);
// CHECK: elapsed_time = (node.end->get_profiling_info<sycl::info::event_profiling::command_end>() - node.start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEventElapsedTime(&elapsed_time, node.start, node.end);
  {
    int errorCode;
  // CHECK: MY_CHECKER((*node.start = q_ct1.ext_oneapi_submit_barrier(), 0));
    MY_CHECKER(hipEventRecord(node.start, 0));
  // CHECK: errorCode = (*node.start = q_ct1.ext_oneapi_submit_barrier(), 0);
    errorCode = hipEventRecord(node.start, 0);
  }

  {
  // CHECK: *node.events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[0]);
  // CHECK: *node.events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[0]);
  // CHECK: *node.events[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[23]);
  // CHECK: *node.events[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(node.events[23]);
  // CHECK: elapsed_time = (node.events[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - node.events[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, node.events[0], node.events[23]);
  }

  {
  // CHECK: **node.ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[0]);
  // CHECK: **node.ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[0]);
  // CHECK: **node.ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[23]);
  // CHECK: **node.ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*node.ev[23]);
  // CHECK: elapsed_time = (*node.ev[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - *node.ev[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, *node.ev[0], *node.ev[23]);
  }

  {
  // CHECK: **(&node)->ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[0]);
  // CHECK: **(&node)->ev[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[0]);
  // CHECK: **(&node)->ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[23]);
  // CHECK: **(&node)->ev[23] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(*(&node)->ev[23]);
  // CHECK:  elapsed_time = (*(&node)->ev[23]->get_profiling_info<sycl::info::event_profiling::command_end>() - *(&node)->ev[0]->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, *(&node)->ev[0], *(&node)->ev[23]);
  }

  {
  // CHECK: *n->p_events[0] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[0]);
  // CHECK: *n->p_events[1] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[1]);
  // CHECK:  *n->p_events[2] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[2]);
  // CHECK:  *n->p_events[3] = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(n->p_events[3]);
  }
}

class C {
// CHECK: dpct::event_ptr start, stop;
  hipEvent_t start, stop;
  float elapsed_time;
  void a() {
  // CHECK: *start = dpct::get_default_queue().ext_oneapi_submit_barrier();
    hipEventRecord(start, 0);
  }
  void b() {
  // CHECK: *stop = dpct::get_default_queue().ext_oneapi_submit_barrier();
    hipEventRecord(stop, 0);
  // CHECK: elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
  }
  void c() {
    hipEventRecord(start, 0);
  }
  void d() {
  // CHECK: *stop = dpct::get_default_queue().ext_oneapi_submit_barrier();
    hipEventRecord(stop, 0);
  // CHECK: elapsed_time = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(&elapsed_time, start, stop);
  }
};

struct S {
  hipEvent_t *events;
};

void foo(int n) {
// CHECK: dpct::event_ptr *events = new dpct::event_ptr[n];
  hipEvent_t *events = new hipEvent_t[n];

// CHECK: *events[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(events[0]);
// CHECK: *events[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(events[1]);
// CHECK: *events[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(events[2]);
// CHECK: *events[3] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(events[3]);

  S s;
// CHECK: *s.events[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s.events[0]);
// CHECK: *s.events[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s.events[1]);
// CHECK: *s.events[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s.events[2]);
// CHECK: *s.events[3] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s.events[3]);

  S *s2 = new S;
// CHECK: *s2->events[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s2->events[0]);
// CHECK: *s2->events[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s2->events[1]);
// CHECK: *s2->events[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s2->events[2]);
// CHECK: *s2->events[3] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(s2->events[3]);
}

void barr(int maxCalls) {
  hipEvent_t evtStart[maxCalls];
  hipEvent_t evtEnd[maxCalls];
  float time[maxCalls];
  for (int i = 0; i < maxCalls; i++) {
    hipEventCreate( &(evtStart[i]) );
    hipEventCreate( &(evtEnd[i]) );
    time[i] = 0.0;
  }

// CHECK: *evtStart[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[0], 0 );
  kernelFunc<<<1, 1>>>();
// CHECK: *evtEnd[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[0], 0 );

// CHECK: *evtStart[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[1], 0 );
  kernelFunc<<<1, 1>>>();
// CHECK: *evtEnd[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[1], 0 );

// CHECK: *evtStart[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[2], 0 );
  kernelFunc<<<1, 1>>>();
// CHECK: *evtEnd[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[2], 0 );

// CHECK: dev_ct1.queues_wait_and_throw();
  hipDeviceSynchronize();

  float total;
  int i=0;
  hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
  float timesum = 0.0f;
  for (int i = 1; i < maxCalls; i++) {
    hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
    timesum += time[i];
  }
  hipEventElapsedTime( &total, evtStart[1], evtEnd[maxCalls-1]);
}

