#include "hip/hip_runtime.h"
// UNSUPPORTED: system-windows
// RUN: cp -r %S %T && cd %T/soft_link_dir/link && rm -rf test && ln -nfs ../target test
// RUN: dpct  --in-root=%T/soft_link_dir --out-root=%T/out  --cuda-include-path="%cuda-path/include"  --process-all -- -I %T/soft_link_dir/link/test -x cuda --cuda-host-only 
// RUN: FileCheck --input-file %T/out/vector_add_format.dp.cpp --match-full-lines %s
// RUN: FileCheck --input-file %T/out/link/test/test/test.hpp --match-full-lines %T/soft_link_dir/link/test/test/test.hpp
// RUN: %if build_lit %{icpx -c -fsycl -DNO_BUILD_TEST  %T/out/vector_add_format.dp.cpp -o %T/out/vector_add_format.dp.o -I %T/out/link/test %}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include "test/test.hpp"
#define VECTOR_SIZE 256

__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}



int main()
{
  //      CHECK:    dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT:    sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  float *d_A, *d_B, *d_C;


  hipMalloc(&d_A, VECTOR_SIZE * sizeof(float));
  hipMalloc(&d_B, VECTOR_SIZE * sizeof(float));
  hipMalloc(&d_C, VECTOR_SIZE * sizeof(float));


  VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);


  float Result[VECTOR_SIZE] = {};
  hipMemcpy(Result, d_C, VECTOR_SIZE * sizeof(float), hipMemcpyDeviceToHost);

  //      CHECK:  dpct::dpct_free(d_A, q_ct1);
  // CHECK-NEXT:  dpct::dpct_free(d_B, q_ct1);
  // CHECK-NEXT:  dpct::dpct_free(d_C, q_ct1);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  for (int i = 0; i < VECTOR_SIZE; i++) {
    if (i % 16 == 0) {
      printf("\n");
    }
    printf("%f ", Result[i]);
  }

    return 0;
}
