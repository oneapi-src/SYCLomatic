// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none -out-root %T/queue_ctn %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/queue_ctn/queue_ctn.dp.cpp


#include "hip/hip_runtime.h"

void bar();
#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;
__constant__ float constData[123 * 4];
hipStream_t s;

// CHECK: void bar1() {
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar1() {
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar2() {
// CHECK-NEXT: s = dpct::get_current_device().create_queue();
// CHECK-NEXT: }
void bar2() {
  hipStreamCreate(&s);
}
// CHECK: void bar3() {
// CHECK-NEXT: s = dpct::get_current_device().create_queue();
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar3() {
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar4() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar4() {
  hipStreamCreate(&s);
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar5() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double));
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar5() {
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar6() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double));
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar6() {
  hipStreamCreate(&s);
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}

void foo1() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  bar();
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}


void foo2() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}

void foo3() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemset(d_A, 23, size);
}

void foo4() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  bar();
  hipMemset(d_A, 23, size);
}

void foo5() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int Err = (q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: Err = (q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait(), 0);
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  int Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
}

#define CUDA_CALL( call) call

void foo6() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: // call in macro
  // CHECK-NEXT: CUDA_CALL(q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait());
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  // call in macro
  CUDA_CALL(hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost ));
}

template <typename T>
void foo7() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  T* d_A_unresolved, h_A_unresolved;
  // CHECK: q_ct1.memcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE );
  // CHECK-NEXT: q_ct1.memcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE ).wait();
  hipMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, hipMemcpyDeviceToHost );
}


template <typename T>
int writeNStage2DDWT() {
    // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
    T *src;
    // CHECK: src = (T *)sycl::malloc_host(10, q_ct1);
    hipHostMalloc((void **)&src, 10, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);

    // CHECK: q_ct1.memcpy(src, src, 10);
    // CHECK-NEXT: q_ct1.memcpy(src, src, 10);
    // CHECK-NEXT: q_ct1.memcpy(src, src, 10).wait();
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);

    return 0;
}
template int writeNStage2DDWT<float>();
template int writeNStage2DDWT<int>();

