// UNSUPPORTED: system-windows
// RUN: dpct --format-range=none -out-root %T/queue_ctn %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/queue_ctn/queue_ctn.dp.cpp


#include "hip/hip_runtime.h"

void bar();
#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;
__constant__ float constData[123 * 4];
hipStream_t s;

// CHECK: void bar1() {
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar1() {
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar2() {
// CHECK-NEXT: s = dpct::get_current_device().create_queue();
// CHECK-NEXT: }
void bar2() {
  hipStreamCreate(&s);
}
// CHECK: void bar3() {
// CHECK-NEXT: s = dpct::get_current_device().create_queue();
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar3() {
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar4() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: dpct::get_default_queue().memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar4() {
  hipStreamCreate(&s);
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar5() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double));
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar5() {
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}
// CHECK: void bar6() {
// CHECK-NEXT: dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: s = dev_ct1.create_queue();
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double));
// CHECK-NEXT: q_ct1.memcpy(d_A, h_A, sizeof(double)).wait();
// CHECK-NEXT: }
void bar6() {
  hipStreamCreate(&s);
  hipStreamCreate(&s);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
  hipMemcpy(d_A, h_A, sizeof(double), hipMemcpyDeviceToHost);
}

__global__ void kernel(float *a, float *b, float *c){
  int i = threadIdx.x;
  c[i] = a[i] + b[i];
}

void bar7(){
// CHECK: float *A, *B, *C;
// CHECK: A = sycl::malloc_device<float>(100, q_ct1);
// CHECK: B = sycl::malloc_device<float>(100, q_ct1);
// CHECK: C = sycl::malloc_device<float>(100, q_ct1);
// CHECK: q_ct1.memcpy(A, h_A, 100 * sizeof(float));
// CHECK: q_ct1.memcpy(B, h_A, 100 * sizeof(float));
  float *A, *B, *C;
  hipMalloc(&A, 100 * sizeof(float));
  hipMalloc(&B, 100 * sizeof(float));
  hipMalloc(&C, 100 * sizeof(float));
  hipMemcpy(A, h_A, 100 * sizeof(float), hipMemcpyDeviceToHost);
  hipMemcpy(B, h_A, 100 * sizeof(float), hipMemcpyDeviceToHost);
  kernel<<<1, 100>>>(A, B, C);
// CHECK: q_ct1.memcpy(h_A, C, 100 * sizeof(float)).wait();
  hipMemcpy(h_A, C, 100 * sizeof(float), hipMemcpyHostToDevice);
}

void foo1() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  bar();
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}


void foo2() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}

void foo3() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemset(d_A, 23, size);
}

void foo4() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  // CHECK-NEXT: bar();
  // CHECK-NEXT: q_ct1.memset(d_A, 23, size).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  bar();
  hipMemset(d_A, 23, size);
}

void foo5() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int Err = (q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: Err = (q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait(), 0);
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  int Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  Err = hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
}

#define CUDA_CALL( call) call

void foo6() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
  // CHECK-NEXT: // call in macro
  // CHECK-NEXT: CUDA_CALL(q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait());
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  // call in macro
  CUDA_CALL(hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost ));
}

template <typename T>
void foo7() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  T* d_A_unresolved, h_A_unresolved;
  // CHECK: q_ct1.memcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE );
  // CHECK-NEXT: q_ct1.memcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE ).wait();
  hipMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A_unresolved, h_A_unresolved, sizeof(T)*SIZE*SIZE, hipMemcpyDeviceToHost );
}


template <typename T>
int writeNStage2DDWT() {
    // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
    // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
    T *src;
    // CHECK: src = (T *)sycl::malloc_host(10, q_ct1);
    hipHostMalloc((void **)&src, 10, hipHostMallocDefault, hipHostMallocDefault, hipHostMallocDefault);

    // CHECK: q_ct1.memcpy(src, src, 10);
    // CHECK-NEXT: q_ct1.memcpy(src, src, 10);
    // CHECK-NEXT: q_ct1.memcpy(src, src, 10).wait();
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);
    hipMemcpy(src, src, 10, hipMemcpyHostToDevice);

    return 0;
}
template int writeNStage2DDWT<float>();
template int writeNStage2DDWT<int>();

