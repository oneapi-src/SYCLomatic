// RUN: cat %s > %T/cuda-device-api.cu
// RUN: cd %T
// RUN: dpct -out-root %T/cuda-device-api cuda-device-api.cu --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-device-api/cuda-device-api.dp.cpp --match-full-lines cuda-device-api.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/cuda-device-api/cuda-device-api.dp.cpp -o %T/cuda-device-api/cuda-device-api.dp.o %}

void foo() {
  size_t *pValue;
  hipLimit_t limit;
  hipSharedMemConfig config;
  unsigned flags;
  int peerDevice;
  int *canAccessPeer;
  int device;
  hipIpcEventHandle_t *handleEvent;
  hipEvent_t event;
  hipIpcMemHandle_t *handleMem;
  void *devPtr;

  // CHECK: /*
  // CHECK-NEXT: DPCT1029:0: SYCL currently does not support getting device resource limits.
  // CHECK-NEXT: The output parameter(s) are set to 0.
  // CHECK-NEXT: */
  // CHECK-NEXT: *pValue = 0;
  hipDeviceGetLimit(pValue, limit);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceSetSharedMemConfig was removed because SYCL
  // CHECK-NEXT: currently does not support configuring shared memory on devices.
  // CHECK-NEXT:*/
  hipDeviceSetSharedMemConfig(config);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipSetDeviceFlags was removed because SYCL currently
  // CHECK-NEXT: does not support setting flags for devices.
  // CHECK-NEXT: */
  hipSetDeviceFlags(flags);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceEnablePeerAccess was removed because SYCL
  // CHECK-NEXT: currently does not support memory access across peer devices.
  // CHECK-NEXT: */
  hipDeviceEnablePeerAccess(peerDevice, flags);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceDisablePeerAccess was removed because SYCL
  // CHECK-NEXT: currently does not support memory access across peer devices.
  // CHECK-NEXT: */
  hipDeviceDisablePeerAccess(peerDevice);

  // CHECK:      /*
  // CHECK-NEXT: DPCT1031:{{[0-9]+}}: Memory accessing across peer devices is an implementation-specific
  // CHECK-NEXT: feature which may not be supported by some SYCL backends and compilers. The
  // CHECK-NEXT: output parameter(s) are set to 0.
  // CHECK-NEXT: */

  // CHECK-NEXT: *canAccessPeer = 0;
  hipDeviceCanAccessPeer(canAccessPeer, device, peerDevice);

  // CHECK: /*
  // CHECK-NEXT: DPCT1030:{{[0-9]+}}: SYCL currently does not support inter-process communication (IPC)
  // CHECK-NEXT: operations. You may need to rewrite the code.
  // CHECK-NEXT: */

  hipIpcGetEventHandle(handleEvent, event);

  // CHECK: /*
  // CHECK-NEXT: DPCT1030:{{[0-9]+}}: SYCL currently does not support inter-process communication (IPC)
  // CHECK-NEXT: operations. You may need to rewrite the code.
  // CHECK-NEXT: */
  hipIpcOpenEventHandle(&event, *handleEvent);

  // CHECK: /*
  // CHECK-NEXT: DPCT1030:{{[0-9]+}}: SYCL currently does not support inter-process communication (IPC)
  // CHECK-NEXT: operations. You may need to rewrite the code.
  // CHECK-NEXT: */
  hipIpcGetMemHandle(handleMem, devPtr);

  // CHECK: /*
  // CHECK-NEXT: DPCT1030:{{[0-9]+}}: SYCL currently does not support inter-process communication (IPC)
  // CHECK-NEXT: operations. You may need to rewrite the code.
  // CHECK-NEXT: */
  hipIpcOpenMemHandle(&devPtr, *handleMem, flags);

  // CHECK: /*
  // CHECK-NEXT: DPCT1030:{{[0-9]+}}: SYCL currently does not support inter-process communication (IPC)
  // CHECK-NEXT: operations. You may need to rewrite the code.
  // CHECK-NEXT: */
  hipIpcCloseMemHandle(devPtr);


  hipFuncCache_t fconfig;

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipFuncSetSharedMemConfig was removed because SYCL
  // CHECK-NEXT: currently does not support configuring shared memory on devices.
  hipFuncSetSharedMemConfig(reinterpret_cast<const void*>(NULL), config );

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipFuncSetCacheConfig was removed because SYCL
  // CHECK-NEXT: currently does not support configuring shared memory on devices.
  hipFuncSetCacheConfig(reinterpret_cast<const void*>(NULL), fconfig);

}
