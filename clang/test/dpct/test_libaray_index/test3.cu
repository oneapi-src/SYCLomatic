// RUN: dpct --format-range=none -out-root %T/output %S/test1.cu %S/test2.cu %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/output/test3.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/output/test3.dp.cpp -o %T/output/test3.dp.o %}


#include <hip/hip_runtime.h>
#include "hipblas.h"

int foo () {
  hipblasStatus_t s;
  hipblasHandle_t handle;
  int N = 275;
  float *x1;
  int *result;

  //CHECK:[&]() {
  //CHECK-NEXT:dpct::blas::out_mem_int_t res(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(handle->get_queue(), N, x1, N, res.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  hipblasIsamax(handle, N, x1, N, result);
}
