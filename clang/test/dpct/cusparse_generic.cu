// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.4, v10.1, v10.2, v11.0, v11.1, v11.2
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2
// RUN: dpct --format-range=none --out-root %T/cusparse_generic %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cusparse_generic/cusparse_generic.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hipsparse.h"

int main() {
  //CHECK:dpct::sparse::sparse_matrix_desc_t spMatDescr;
  //CHECK-NEXT:int64_t rows;
  //CHECK-NEXT:int64_t cols;
  //CHECK-NEXT:int64_t nnz;
  //CHECK-NEXT:void *csrRowOffsets;
  //CHECK-NEXT:void *csrColInd;
  //CHECK-NEXT:void *csrValues;
  //CHECK-NEXT:dpct::library_data_t csrRowOffsetsType;
  //CHECK-NEXT:dpct::library_data_t csrColIndType;
  //CHECK-NEXT:oneapi::mkl::index_base idxBase;
  //CHECK-NEXT:dpct::library_data_t valueType;
  //CHECK-NEXT:dpct::sparse::matrix_format format;
  hipsparseSpMatDescr_t spMatDescr;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  void *csrRowOffsets;
  void *csrColInd;
  void *csrValues;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexBase_t idxBase;
  hipDataType valueType;
  hipsparseFormat_t format;

  //CHECK:spMatDescr = std::make_shared<dpct::sparse::sparse_matrix_desc>(rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, idxBase, valueType, dpct::sparse::matrix_format::csr);
  //CHECK-NEXT:spMatDescr.reset();
  //CHECK-NEXT:spMatDescr->get_desc(&rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
  //CHECK-NEXT:spMatDescr->get_format(&format);
  //CHECK-NEXT:spMatDescr->get_base(&idxBase);
  //CHECK-NEXT:spMatDescr->get_value(&csrValues);
  //CHECK-NEXT:spMatDescr->set_value(csrValues);
  //CHECK-NEXT:spMatDescr->set_pointers(csrRowOffsets, csrColInd, csrValues);
  //CHECK-NEXT:spMatDescr->get_size(&rows, &cols, &nnz);
  hipsparseCreateCsr(&spMatDescr, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, idxBase, valueType);
  hipsparseDestroySpMat(spMatDescr);
  hipsparseCsrGet(spMatDescr, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
  hipsparseSpMatGetFormat(spMatDescr, &format);
  hipsparseSpMatGetIndexBase(spMatDescr, &idxBase);
  hipsparseSpMatGetValues(spMatDescr, &csrValues);
  hipsparseSpMatSetValues(spMatDescr, csrValues);
  hipsparseCsrSetPointers(spMatDescr, csrRowOffsets, csrColInd, csrValues);
  hipsparseSpMatGetSize(spMatDescr, &rows, &cols, &nnz);

  //CHECK:void *data;
  //CHECK-NEXT:spMatDescr->get_attribute(dpct::sparse::matrix_attribute::uplo, &data, sizeof(oneapi::mkl::uplo));
  //CHECK-NEXT:spMatDescr->set_attribute(dpct::sparse::matrix_attribute::diag, data, sizeof(oneapi::mkl::diag));
  void *data;
  hipsparseSpMatGetAttribute(spMatDescr, HIPSPARSE_SPMAT_FILL_MODE, &data, sizeof(hipsparseFillMode_t));
  hipsparseSpMatSetAttribute(spMatDescr, HIPSPARSE_SPMAT_DIAG_TYPE, data, sizeof(hipsparseDiagType_t));

  //CHECK:std::shared_ptr<dpct::sparse::dense_matrix_desc> dnMatDescr;
  //CHECK-NEXT:int64_t ld;
  //CHECK-NEXT:oneapi::mkl::layout order;
  //CHECK-NEXT:void *values;
  hipsparseDnMatDescr_t dnMatDescr;
  int64_t ld;
  hipsparseOrder_t order;
  void *values;

  //CHECK:dnMatDescr = std::make_shared<dpct::sparse::dense_matrix_desc>(rows, cols, ld, values, valueType, order);
  //CHECK-NEXT:dnMatDescr.reset();
  //CHECK-NEXT:dnMatDescr->get_desc(&rows, &cols, &ld, &values, &valueType, &order);
  //CHECK-NEXT:dnMatDescr->get_value(&values);
  //CHECK-NEXT:dnMatDescr->set_value(values);
  hipsparseCreateDnMat(&dnMatDescr, rows, cols, ld, values, valueType, order);
  hipsparseDestroyDnMat(dnMatDescr);
  hipsparseDnMatGet(dnMatDescr, &rows, &cols, &ld, &values, &valueType, &order);
  hipsparseDnMatGetValues(dnMatDescr, &values);
  hipsparseDnMatSetValues(dnMatDescr, values);

  //CHECK:std::shared_ptr<dpct::sparse::dense_vector_desc> dnVecDescr;
  //CHECK-NEXT:int64_t size;
  hipsparseDnVecDescr_t dnVecDescr;
  int64_t size;

  //CHECK:sycl::queue* handle;
  //CHECK-NEXT:const void *alpha;
  //CHECK-NEXT:const void *beta;
  //CHECK-NEXT:dpct::sparse::sparse_matrix_desc_t matA;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_matrix_desc> matB;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_matrix_desc> matC;
  //CHECK-NEXT:dpct::library_data_t computeType;
  //CHECK-NEXT:int alg1;
  //CHECK-NEXT:size_t bufferSize;
  //CHECK-NEXT:void *externalBuffer;
  //CHECK-NEXT:bufferSize = 0;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpMM_preprocess was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  //CHECK-NEXT:dpct::sparse::spmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, alpha, matA, matB, beta, matC, computeType);
  hipsparseHandle_t handle;
  const void *alpha;
  const void *beta;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB;
  hipsparseDnMatDescr_t matC;
  hipDataType computeType;
  hipsparseSpMMAlg_t alg1;
  size_t bufferSize;
  void *externalBuffer;
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, &bufferSize);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, externalBuffer);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, externalBuffer);

  //CHECK:std::shared_ptr<dpct::sparse::dense_vector_desc> vecX;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_vector_desc> vecY;
  //CHECK-NEXT:int alg2;
  //CHECK-NEXT:bufferSize = 0;
  //CHECK-NEXT:dpct::sparse::spmv(*handle, oneapi::mkl::transpose::nontrans, alpha, matA, vecX, beta, vecY, computeType);
  hipsparseConstDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseSpMVAlg_t alg2;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecX, beta, vecY, computeType, alg2, &bufferSize);
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecX, beta, vecY, computeType, alg2, externalBuffer);

  //CHECK:dnVecDescr = std::make_shared<dpct::sparse::dense_vector_desc>(size, values, valueType);
  //CHECK-NEXT:dnVecDescr.reset();
  //CHECK-NEXT:dnVecDescr->get_desc(&size, &values, &valueType);
  //CHECK-NEXT:dnVecDescr->get_value(&values);
  //CHECK-NEXT:dnVecDescr->set_value(values);
  hipsparseCreateDnVec(&dnVecDescr, size, values, valueType);
  hipsparseDestroyDnVec(dnVecDescr);
  hipsparseDnVecGet(dnVecDescr, &size, &values, &valueType);
  hipsparseDnVecGetValues(dnVecDescr, &values);
  hipsparseDnVecSetValues(dnVecDescr, values);
  return 0;
}
