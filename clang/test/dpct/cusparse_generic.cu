// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.4, v10.1, v10.2, v11.0, v11.1, v11.2
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2
// RUN: dpct --format-range=none --out-root %T/cusparse_generic %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cusparse_generic/cusparse_generic.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hipsparse.h"

int main() {
  //CHECK:dpct::sparse::sparse_matrix_desc_t spMatDescr;
  //CHECK-NEXT:int64_t rows;
  //CHECK-NEXT:int64_t cols;
  //CHECK-NEXT:int64_t nnz;
  //CHECK-NEXT:void *csrRowOffsets;
  //CHECK-NEXT:void *csrColInd;
  //CHECK-NEXT:void *csrValues;
  //CHECK-NEXT:dpct::library_data_t csrRowOffsetsType;
  //CHECK-NEXT:dpct::library_data_t csrColIndType;
  //CHECK-NEXT:oneapi::mkl::index_base idxBase;
  //CHECK-NEXT:dpct::library_data_t valueType;
  //CHECK-NEXT:dpct::sparse::matrix_format format;
  hipsparseSpMatDescr_t spMatDescr;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  void *csrRowOffsets;
  void *csrColInd;
  void *csrValues;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexBase_t idxBase;
  hipDataType valueType;
  hipsparseFormat_t format;

  //CHECK:spMatDescr = std::make_shared<dpct::sparse::sparse_matrix_desc>(rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, idxBase, valueType, dpct::sparse::matrix_format::csr);
  //CHECK-NEXT:spMatDescr.reset();
  //CHECK-NEXT:spMatDescr->get_desc(&rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
  //CHECK-NEXT:spMatDescr->get_format(&format);
  //CHECK-NEXT:spMatDescr->get_base(&idxBase);
  //CHECK-NEXT:spMatDescr->get_value(&csrValues);
  //CHECK-NEXT:spMatDescr->set_value(csrValues);
  //CHECK-NEXT:spMatDescr->set_pointers(csrRowOffsets, csrColInd, csrValues);
  //CHECK-NEXT:spMatDescr->get_size(&rows, &cols, &nnz);
  hipsparseCreateCsr(&spMatDescr, rows, cols, nnz, csrRowOffsets, csrColInd, csrValues, csrRowOffsetsType, csrColIndType, idxBase, valueType);
  hipsparseDestroySpMat(spMatDescr);
  hipsparseCsrGet(spMatDescr, &rows, &cols, &nnz, &csrRowOffsets, &csrColInd, &csrValues, &csrRowOffsetsType, &csrColIndType, &idxBase, &valueType);
  hipsparseSpMatGetFormat(spMatDescr, &format);
  hipsparseSpMatGetIndexBase(spMatDescr, &idxBase);
  hipsparseSpMatGetValues(spMatDescr, &csrValues);
  hipsparseSpMatSetValues(spMatDescr, csrValues);
  hipsparseCsrSetPointers(spMatDescr, csrRowOffsets, csrColInd, csrValues);
  hipsparseSpMatGetSize(spMatDescr, &rows, &cols, &nnz);

  //CHECK:void *data;
  //CHECK-NEXT:spMatDescr->get_attribute(dpct::sparse::matrix_attribute::uplo, &data, sizeof(oneapi::mkl::uplo));
  //CHECK-NEXT:spMatDescr->set_attribute(dpct::sparse::matrix_attribute::diag, data, sizeof(oneapi::mkl::diag));
  void *data;
  hipsparseSpMatGetAttribute(spMatDescr, HIPSPARSE_SPMAT_FILL_MODE, &data, sizeof(hipsparseFillMode_t));
  hipsparseSpMatSetAttribute(spMatDescr, HIPSPARSE_SPMAT_DIAG_TYPE, data, sizeof(hipsparseDiagType_t));

  //CHECK:std::shared_ptr<dpct::sparse::dense_matrix_desc> dnMatDescr;
  //CHECK-NEXT:int64_t ld;
  //CHECK-NEXT:oneapi::mkl::layout order;
  //CHECK-NEXT:void *values;
  hipsparseDnMatDescr_t dnMatDescr;
  int64_t ld;
  hipsparseOrder_t order;
  void *values;

  //CHECK:dnMatDescr = std::make_shared<dpct::sparse::dense_matrix_desc>(rows, cols, ld, values, valueType, order);
  //CHECK-NEXT:dnMatDescr.reset();
  //CHECK-NEXT:dnMatDescr->get_desc(&rows, &cols, &ld, &values, &valueType, &order);
  //CHECK-NEXT:values = dnMatDescr->get_value();
  //CHECK-NEXT:dnMatDescr->set_value(values);
  hipsparseCreateDnMat(&dnMatDescr, rows, cols, ld, values, valueType, order);
  hipsparseDestroyDnMat(dnMatDescr);
  hipsparseDnMatGet(dnMatDescr, &rows, &cols, &ld, &values, &valueType, &order);
  hipsparseDnMatGetValues(dnMatDescr, &values);
  hipsparseDnMatSetValues(dnMatDescr, values);

  //CHECK:std::shared_ptr<dpct::sparse::dense_vector_desc> dnVecDescr;
  //CHECK-NEXT:int64_t size;
  hipsparseDnVecDescr_t dnVecDescr;
  int64_t size;

  //CHECK:sycl::queue* handle;
  //CHECK-NEXT:const void *alpha;
  //CHECK-NEXT:const void *beta;
  //CHECK-NEXT:dpct::sparse::sparse_matrix_desc_t matA;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_matrix_desc> matB;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_matrix_desc> matC;
  //CHECK-NEXT:dpct::library_data_t computeType;
  //CHECK-NEXT:int alg1;
  //CHECK-NEXT:size_t bufferSize;
  //CHECK-NEXT:void *externalBuffer;
  //CHECK-NEXT:bufferSize = 0;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpMM_preprocess was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  //CHECK-NEXT:dpct::sparse::spmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, alpha, matA, matB, beta, matC, computeType);
  hipsparseHandle_t handle;
  const void *alpha;
  const void *beta;
  hipsparseSpMatDescr_t matA;
  hipsparseDnMatDescr_t matB;
  hipsparseDnMatDescr_t matC;
  hipDataType computeType;
  hipsparseSpMMAlg_t alg1;
  size_t bufferSize;
  void *externalBuffer;
  hipsparseSpMM_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, &bufferSize);
  hipsparseSpMM_preprocess(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, externalBuffer);
  hipsparseSpMM(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, matB, beta, matC, computeType, alg1, externalBuffer);

  //CHECK:std::shared_ptr<dpct::sparse::dense_vector_desc> vecX;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_vector_desc> vecY;
  //CHECK-NEXT:int alg2;
  //CHECK-NEXT:bufferSize = 0;
  //CHECK-NEXT:dpct::sparse::spmv(*handle, oneapi::mkl::transpose::nontrans, alpha, matA, vecX, beta, vecY, computeType);
  hipsparseConstDnVecDescr_t vecX;
  hipsparseDnVecDescr_t vecY;
  hipsparseSpMVAlg_t alg2;
  hipsparseSpMV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecX, beta, vecY, computeType, alg2, &bufferSize);
  hipsparseSpMV(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA, vecX, beta, vecY, computeType, alg2, externalBuffer);

  //CHECK:dnVecDescr = std::make_shared<dpct::sparse::dense_vector_desc>(size, values, valueType);
  //CHECK-NEXT:dnVecDescr.reset();
  //CHECK-NEXT:dnVecDescr->get_desc(&size, &values, &valueType);
  //CHECK-NEXT:values = dnVecDescr->get_value();
  //CHECK-NEXT:dnVecDescr->set_value(values);
  hipsparseCreateDnVec(&dnVecDescr, size, values, valueType);
  hipsparseDestroyDnVec(dnVecDescr);
  hipsparseDnVecGet(dnVecDescr, &size, &values, &valueType);
  hipsparseDnVecGetValues(dnVecDescr, &values);
  hipsparseDnVecSetValues(dnVecDescr, values);

  //CHECK:oneapi::mkl::sparse::matmat_descr_t descr;
  //CHECK-NEXT:dpct::sparse::sparse_matrix_desc_t matA_sparse;
  //CHECK-NEXT:dpct::sparse::sparse_matrix_desc_t matB_sparse;
  //CHECK-NEXT:dpct::sparse::sparse_matrix_desc_t matC_sparse;
  //CHECK-NEXT:int alg3;
  //CHECK-NEXT:size_t workspace_size;
  //CHECK-NEXT:void *workspace;
  //CHECK-NEXT:oneapi::mkl::sparse::init_matmat_descr(&descr);
  //CHECK-NEXT:dpct::sparse::spgemm_work_estimation(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, alpha, matA_sparse, matB_sparse, beta, matC_sparse, descr, &workspace_size, workspace);
  //CHECK-NEXT:dpct::sparse::spgemm_compute(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, alpha, matA_sparse, matB_sparse, beta, matC_sparse, descr, &workspace_size, workspace);
  //CHECK-NEXT:dpct::sparse::spgemm_finalize(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, alpha, matA_sparse, matB_sparse, beta, matC_sparse, descr);
  //CHECK-NEXT:oneapi::mkl::sparse::release_matmat_descr(&descr);
  hipsparseSpGEMMDescr_t descr;
  hipsparseSpMatDescr_t matA_sparse;
  hipsparseSpMatDescr_t matB_sparse;
  hipsparseSpMatDescr_t matC_sparse;
  hipsparseSpGEMMAlg_t alg3;
  size_t workspace_size;
  void *workspace;
  hipsparseSpGEMM_createDescr(&descr);
  hipsparseSpGEMM_workEstimation(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, matB_sparse, beta, matC_sparse, computeType, alg3, descr, &workspace_size, workspace);
  hipsparseSpGEMM_compute(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, matB_sparse, beta, matC_sparse, computeType, alg3, descr, &workspace_size, workspace);
  hipsparseSpGEMM_copy(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, matB_sparse, beta, matC_sparse, computeType, alg3, descr);
  hipsparseSpGEMM_destroyDescr(descr);

  //CHECK:int spsvDescr;
  //CHECK-NEXT:int alg4;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpSV_createDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpSV_bufferSize was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  //CHECK-NEXT:dpct::sparse::spsv_optimize(*handle, oneapi::mkl::transpose::nontrans, matA_sparse);
  //CHECK-NEXT:dpct::sparse::spsv(*handle, oneapi::mkl::transpose::nontrans, alpha, matA_sparse, vecX, vecY, computeType);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpSV_destroyDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT:*/
  hipsparseSpSVDescr_t spsvDescr;
  hipsparseSpSVAlg_t alg4;
  hipsparseSpSV_createDescr(&spsvDescr);
  hipsparseSpSV_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, vecX, vecY, computeType, alg4, spsvDescr, &workspace_size);
  hipsparseSpSV_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, vecX, vecY, computeType, alg4, spsvDescr, workspace);
  hipsparseSpSV_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, alpha, matA_sparse, vecX, vecY, computeType, alg4, spsvDescr);
  hipsparseSpSV_destroyDescr(spsvDescr);

  return 0;
}

void foo1() {
  hipsparseSpMatDescr_t spMatDescr;
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  void *csrRowOffsets;
  void *csrColInd;
  void *csrValues;
  hipsparseIndexType_t csrRowOffsetsType;
  hipsparseIndexType_t csrColIndType;
  hipsparseIndexBase_t idxBase;
  hipDataType valueType;

  //CHECK:/*
  //CHECK-NEXT:DPCT1119:{{[0-9]+}}: The memory of row_ptr is not allocated. You need to allocate the memory before invoking oneapi::mkl::sparse::set_csr_data().
  //CHECK-NEXT:*/
  //CHECK-NEXT:spMatDescr = std::make_shared<dpct::sparse::sparse_matrix_desc>(rows, cols, nnz, nullptr, nullptr, nullptr, csrRowOffsetsType, csrColIndType, idxBase, valueType, dpct::sparse::matrix_format::csr);
  hipsparseCreateCsr(&spMatDescr, rows, cols, nnz, NULL, NULL, NULL, csrRowOffsetsType, csrColIndType, idxBase, valueType);
}

//CHECK:void foo2(oneapi::mkl::sparse::matmat_descr_t *descr) {
//CHECK-NEXT:  oneapi::mkl::sparse::release_matmat_descr(descr);
void foo2(hipsparseSpGEMMDescr_t *descr) {
  hipsparseSpGEMM_destroyDescr(*descr);
}

//CHECK:oneapi::mkl::sparse::matmat_descr_t foo3();
//CHECK-NEXT:void foo4() {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1120:{{[0-9]+}}: Taking the address of an rvalue "foo3()" is not allowed. You need to adjust the code.
//CHECK-NEXT:  */
//CHECK-NEXT:  oneapi::mkl::sparse::release_matmat_descr(&foo3());
hipsparseSpGEMMDescr_t foo3();
void foo4() {
  hipsparseSpGEMM_destroyDescr(foo3());
}
