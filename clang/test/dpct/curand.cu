// CHECKME
// RUN: cat %s > %T/curand.cu
// RUN: cd %T
//RUN: dpct -out-root %T/curand curand.cu --usm-level=none --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
//RUN: FileCheck --input-file %T/curand/curand.dp.cpp --match-full-lines curand.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/curand/curand.dp.cpp -o %T/curand/curand.dp.o %}
//CHECK:// CHECKME
//CHECK:#include <sycl/sycl.hpp>
//CHECK:#include <dpct/dpct.hpp>
//CHECK:#include <dpct/rng_utils.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

int main(){
  //CHECK:int s1;
  //CHECK-NEXT:int s2;
  //CHECK-NEXT:dpct::rng::host_rng_ptr rng;
  //CHECK-NEXT:rng =
  //CHECK-NEXT:    dpct::rng::create_host_rng(dpct::rng::random_engine_type::philox4x32x10);
  //CHECK-NEXT:rng->set_seed(1337ull);
  //CHECK-NEXT:float *d_data;
  hiprandStatus_t s1;
  hiprandStatus s2;
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
  float *d_data;

  //CHECK:rng->generate_uniform(d_data, 100 * 100);
  hiprandGenerateUniform(rng, d_data, 100*100);

  //CHECK:s1 = DPCT_CHECK_ERROR(rng->generate_uniform(d_data, 100 * 100));
  s1 = hiprandGenerateUniform(rng, d_data, 100*100);

  //CHECK:s1 = DPCT_CHECK_ERROR(rng->generate_lognormal(d_data, 100 * 100, 123, 456));
  s1 = hiprandGenerateLogNormal(rng, d_data, 100*100, 123, 456);

  //CHECK:s1 = DPCT_CHECK_ERROR(rng->generate_gaussian(d_data, 100 * 100, 123, 456));
  s1 = hiprandGenerateNormal(rng, d_data, 100*100, 123, 456);

  double* d_data_d;
  //CHECK:rng->generate_uniform(d_data_d, 100 * 100);
  //CHECK-NEXT:rng->generate_lognormal(d_data_d, 100 * 100, 123, 456);
  //CHECK-NEXT:rng->generate_gaussian(d_data_d, 100 * 100, 123, 456);
  hiprandGenerateUniformDouble(rng, d_data_d, 100*100);
  hiprandGenerateLogNormalDouble(rng, d_data_d, 100*100, 123, 456);
  hiprandGenerateNormalDouble(rng, d_data_d, 100*100, 123, 456);

  unsigned int* d_data_ui;
  //CHECK:s1 = DPCT_CHECK_ERROR(rng->generate_uniform_bits(d_data_ui, 100 * 100));
  s1 = hiprandGenerate(rng, d_data_ui, 100*100);

  //CHECK:s1 = DPCT_CHECK_ERROR(rng->generate_poisson(d_data_ui, 100 * 100, 123.456));
  s1 = hiprandGeneratePoisson(rng, d_data_ui, 100*100, 123.456);

  unsigned long long* d_data_ull;
  //CHECK:rng->generate_uniform_bits(d_data_ull, 100 * 100);
  hiprandGenerateLongLong(rng, d_data_ull, 100*100);

  //CHECK: if (s1 = DPCT_CHECK_ERROR(
  //CHECK-NEXT:   rng->generate_uniform_bits(d_data_ull, 100 * 100))) {}
  if(s1 = hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  //CHECK:if (DPCT_CHECK_ERROR(rng->generate_uniform_bits(d_data_ull, 100 * 100))) {}
  if(hiprandGenerateLongLong(rng, d_data_ull, 100*100)){}

  //CHECK:dpct::rng::host_rng_ptr rng2;
  //CHECK-NEXT:rng2 = dpct::rng::create_host_rng(dpct::rng::random_engine_type::sobol);
  //CHECK-NEXT:rng2->set_dimensions(1111);
  //CHECK-NEXT:rng2->generate_uniform(d_data, 100 * 100);
  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  hiprandGenerateUniform(rng2, d_data, 100*100);

  //CHECK:rng->skip_ahead(100);
  //CHECK-NEXT:s1 = DPCT_CHECK_ERROR(rng2->skip_ahead(200));
  hiprandSetGeneratorOffset(rng, 100);
  s1 = hiprandSetGeneratorOffset(rng2, 200);

  //CHECK:rng.reset();
  //CHECK-NEXT:s1 = DPCT_CHECK_ERROR(rng.reset());
  hiprandDestroyGenerator(rng);
  s1 = hiprandDestroyGenerator(rng);
}

//CHECK:int foo1();
hiprandStatus_t foo1();
//CHECK:int foo2();
hiprandStatus foo2();

//CHECK:class A{
//CHECK-NEXT:public:
//CHECK-NEXT:  A(){
//CHECK-NEXT:    rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::sobol);
//CHECK-NEXT:    rng->set_dimensions(1243);
//CHECK-NEXT:  }
//CHECK-NEXT:  ~A(){
//CHECK-NEXT:    rng.reset();
//CHECK-NEXT:  }
//CHECK-NEXT:private:
//CHECK-NEXT:  dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:};
class A{
public:
  A(){
    hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_DEFAULT);
    hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
  }
  ~A(){
    hiprandDestroyGenerator(rng);
  }
private:
  hiprandGenerator_t rng;
};



void bar1(){
//CHECK:dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1032:{{[0-9]+}}: A different random number generator is used. You may need to
//CHECK-NEXT:adjust the code.
//CHECK-NEXT:*/
//CHECK-NEXT:rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::mcg59);
//CHECK-NEXT:rng->set_seed(1337ull);
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_XORWOW);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
}


void bar2(){
//CHECK:dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1032:{{[0-9]+}}: A different random number generator is used. You may need to
//CHECK-NEXT:adjust the code.
//CHECK-NEXT:*/
//CHECK-NEXT:rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::sobol);
//CHECK-NEXT:rng->set_dimensions(1243);
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
//CHECK:if (stat != 0) {
  if (stat != HIPRAND_STATUS_SUCCESS) {
    fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
  }
}

void bar3(){
//CHECK:dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:curandErrCheck(
//CHECK-NEXT:    DPCT_CHECK_ERROR(rng = dpct::rng::create_host_rng(
//CHECK-NEXT:                         dpct::rng::random_engine_type::philox4x32x10)));
//CHECK-NEXT:curandErrCheck(DPCT_CHECK_ERROR(rng->set_seed(1337ull)));
//CHECK-NEXT:float *d_data;
//CHECK-NEXT:curandErrCheck(DPCT_CHECK_ERROR(rng->generate_uniform(d_data, 100 * 100)));
//CHECK-NEXT:curandErrCheck(DPCT_CHECK_ERROR(rng.reset()));
  hiprandGenerator_t rng;
  curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10));
  curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
  float *d_data;
  curandErrCheck(hiprandGenerateUniform(rng, d_data, 100*100));
  curandErrCheck(hiprandDestroyGenerator(rng));
}

void bar4(){
//CHECK:dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1033:{{[0-9]+}}: Migrated code uses a basic Sobol generator. Initialize
//CHECK-NEXT:oneapi::mkl::rng::sobol generator with user-defined direction numbers to use
//CHECK-NEXT:it as Scrambled Sobol generator.
//CHECK-NEXT:*/
//CHECK-NEXT:rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::sobol);
//CHECK-NEXT:rng->set_dimensions(1243);
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32);
  hiprandSetQuasiRandomGeneratorDimensions(rng, 1243);
}

void bar5(){
//CHECK:dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:rng = dpct::rng::create_host_rng((dpct::rng::random_engine_type)101);
//CHECK-NEXT:rng->set_seed(1337ull);
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, (hiprandRngType_t)101);
  hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull);
}

//CHECK:int bar6() try {
//CHECK-NEXT:  float *d_data;
//CHECK-NEXT:  dpct::rng::host_rng_ptr rng2;
//CHECK-NEXT:  rng2 = dpct::rng::create_host_rng(dpct::rng::random_engine_type::sobol);
//CHECK-NEXT:  rng2->set_dimensions(1111);
//CHECK:  return DPCT_CHECK_ERROR(rng2->generate_uniform(d_data, 100 * 100));
//CHECK-NEXT:}
int bar6(){
  float *d_data;
  hiprandGenerator_t rng2;
  hiprandCreateGenerator(&rng2, HIPRAND_RNG_QUASI_DEFAULT);
  hiprandSetQuasiRandomGeneratorDimensions(rng2, 1111);
  return hiprandGenerateUniform(rng2, d_data, 100*100);
}

void bar7() {
  hiprandGenerator_t rng;
  // CHECK: dpct::rng::random_engine_type rngT;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT1 = dpct::rng::random_engine_type::mcg59;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1032:{{[0-9]+}}: A different random number generator is used. You may need to
  // CHECK-NEXT: adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::rng::random_engine_type rngT2 = dpct::rng::random_engine_type::mcg59;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT3 = dpct::rng::random_engine_type::mrg32k3a;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT4 = dpct::rng::random_engine_type::mt2203;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT5 = dpct::rng::random_engine_type::mt19937;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT6 =
  // CHECK-NEXT:     dpct::rng::random_engine_type::philox4x32x10;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT7 = dpct::rng::random_engine_type::sobol;
  // CHECK-NEXT: dpct::rng::random_engine_type rngT8 = dpct::rng::random_engine_type::sobol;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1033:{{[0-9]+}}: Migrated code uses a basic Sobol generator. Initialize
  // CHECK-NEXT: oneapi::mkl::rng::sobol generator with user-defined direction numbers to use
  // CHECK-NEXT: it as Scrambled Sobol generator.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::rng::random_engine_type rngT9 = dpct::rng::random_engine_type::sobol;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1032:{{[0-9]+}}: A different random number generator is used. You may need to
  // CHECK-NEXT: adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::rng::random_engine_type rngT10 = dpct::rng::random_engine_type::sobol;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1032:{{[0-9]+}}: A different random number generator is used. You may need to
  // CHECK-NEXT: adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::rng::random_engine_type rngT11 = dpct::rng::random_engine_type::sobol;
  hiprandRngType_t rngT;
  hiprandRngType_t rngT1 = HIPRAND_RNG_PSEUDO_DEFAULT;
  hiprandRngType_t rngT2 = HIPRAND_RNG_PSEUDO_XORWOW;
  hiprandRngType_t rngT3 = HIPRAND_RNG_PSEUDO_MRG32K3A;
  hiprandRngType_t rngT4 = HIPRAND_RNG_PSEUDO_MTGP32;
  hiprandRngType_t rngT5 = HIPRAND_RNG_PSEUDO_MT19937;
  hiprandRngType_t rngT6 = HIPRAND_RNG_PSEUDO_PHILOX4_32_10;
  hiprandRngType_t rngT7 = HIPRAND_RNG_QUASI_DEFAULT;
  hiprandRngType_t rngT8 = HIPRAND_RNG_QUASI_SOBOL32;
  hiprandRngType_t rngT9 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL32;
  hiprandRngType_t rngT10 = HIPRAND_RNG_QUASI_SOBOL64;
  hiprandRngType_t rngT11 = HIPRAND_RNG_QUASI_SCRAMBLED_SOBOL64;
  // CHECK: rng = dpct::rng::create_host_rng<true>(rngT);
  hiprandCreateGeneratorHost(&rng, rngT);
}
