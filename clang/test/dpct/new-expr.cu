// RUN: dpct --format-range=none --usm-level=none -out-root %T/new-expr %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/new-expr/new-expr.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/new-expr/new-expr.dp.cpp -o %T/new-expr/new-expr.dp.o %}

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define NEW_STREAM new dpct::queue_ptr
// CHECK-NEXT: #define NEW_EVENT new dpct::event_ptr
// CHECK-EMPTY:
// CHECK-NEXT: #define NEW(T) new T
#define NEW_STREAM new hipStream_t
#define NEW_EVENT new hipEvent_t

#define NEW(T) new T

void foo() {
  int n = 16;

  // CHECK: dpct::queue_ptr *stream = new dpct::queue_ptr;
  // CHECK-NEXT: stream = new dpct::queue_ptr();
  // CHECK-NEXT: stream = NEW_STREAM;
  // CHECK-NEXT: stream = NEW(dpct::queue_ptr);
  // CHECK-NEXT: dpct::queue_ptr *streams = new dpct::queue_ptr[n];
  hipStream_t *stream = new hipStream_t;
  stream = new hipStream_t();
  stream = NEW_STREAM;
  stream = NEW(hipStream_t);
  hipStream_t *streams = new hipStream_t[n];

  // CHECK: dpct::event_ptr *event = new dpct::event_ptr;
  // CHECK-NEXT: event = new dpct::event_ptr();
  // CHECK-NEXT: event = NEW_EVENT;
  // CHECK-NEXT: event = NEW(dpct::event_ptr);
  // CHECK-NEXT: dpct::event_ptr *events = new dpct::event_ptr[n];
  hipEvent_t *event = new hipEvent_t;
  event = new hipEvent_t();
  event = NEW_EVENT;
  event = NEW(hipEvent_t);
  hipEvent_t *events = new hipEvent_t[n];
}

