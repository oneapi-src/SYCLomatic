// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);

  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hisnan2(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hle2(h2, h2_1);
  h2_2 = __hle2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hleu2(h2, h2);
  __hleu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hlt2(h2, h2_1);
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hltu2(h2, h2);
  __hltu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hne2(h2, h2_1);
  h2_2 = __hne2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hneu2(h2, h2);
  __hneu2(h2, h2);
}

int main() { return 0; }
