// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::ext::intel::math::erfcinv(d0);
  d2 = erfcinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfinv(d0);
  d2 = erfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorm(d0);
  d2 = normcdf(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorminv(d0);
  d2 = normcdfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(i, &d0);
  d2 = norm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(d1, &d0);
  d2 = norm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(i, &d0);
  d2 = rnorm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(d1, &d0);
  d2 = rnorm(d1, &d0);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::ext::intel::math::erfcinv(f0);
  f2 = erfcinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv((float)i);
  f2 = erfcinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::erfinv(f0);
  f2 = erfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfinv((float)i);
  f2 = erfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm(f0);
  f2 = normcdff(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm((float)i);
  f2 = normcdff(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv(f0);
  f2 = normcdfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv((float)i);
  f2 = normcdfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::norm(i, &f0);
  f2 = normf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::norm(f1, &f0);
  f2 = normf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(i, &f0);
  f2 = rnormf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(f1, &f0);
  f2 = rnormf(f1, &f0);
}

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul(h, h_1);
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hneg(h_1);
  h_2 = __hneg(h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub(h, h_1);
  h_2 = __hsub(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hadd(h, h_1);
  h_2 = __hadd(h, h_1);

  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::heq(h, h_1);
  b = __heq(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hge(h, h_1);
  b = __hge(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgt(h, h_1);
  b = __hgt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hle(h, h_1);
  b = __hle(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hlt(h, h_1);
  b = __hlt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hne(h, h_1);
  b = __hne(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::h2div(h2, h2_1);
  h2_2 = __h2div(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2(h2, h2_1);
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2(h2, h2_1);
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hneg2(h2_1);
  h2_2 = __hneg2(h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2(h2, h2_1);
  h2_2 = __hsub2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);

  // CHECK: h2_2 = sycl::ext::intel::math::hisnan2(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hle2(h2, h2_1);
  h2_2 = __hle2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hleu2(h2, h2);
  __hleu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hlt2(h2, h2_1);
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hltu2(h2, h2);
  __hltu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hne2(h2, h2_1);
  h2_2 = __hne2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hneu2(h2, h2);
  __hneu2(h2, h2);
}

__global__ void kernelFuncInt() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  // CHECK: ll = sycl::max<long long>(ll, ll);
  // CHECK-NEXT: ll = sycl::min<long long>(ll, ll);
  // CHECK-NEXT: ull = sycl::max<unsigned long long>(ull, ull);
  // CHECK-NEXT: ull = sycl::min<unsigned long long>(ull, ull);
  // CHECK-NEXT: u = sycl::max<unsigned int>(u, u);
  // CHECK-NEXT: u = sycl::min<unsigned int>(u, u);
  ll = llmax(ll, ll);
  ll = llmin(ll, ll);
  ull = ullmax(ull, ull);
  ull = ullmin(ull, ull);
  u = umax(u, u);
  u = umin(u, u);
}

int main() { return 0; }
