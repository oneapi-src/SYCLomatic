// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hadd(h, h_1);
  h_2 = __hadd(h, h_1);

  // CHECK: h_2 = sycl::ext::intel::math::hmul(h, h_1);
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub(h, h_1);
  h_2 = __hsub(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hneg(h_1);
  h_2 = __hneg(h_1);


  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::heq(h, h_1);
  b = __heq(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hne(h, h_1);
  b = __hne(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hge(h, h_1);
  b = __hge(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgt(h, h_1);
  b = __hgt(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hle(h, h_1);
  b = __hle(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hlt(h, h_1);
  b = __hlt(h, h_1);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);

  // CHECK: h2_2 = sycl::ext::intel::math::hisnan2(h2);
  h2_2 = __hisnan2(h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hle2(h2, h2_1);
  h2_2 = __hle2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hleu2(h2, h2);
  __hleu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hlt2(h2, h2_1);
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hltu2(h2, h2);
  __hltu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hne2(h2, h2_1);
  h2_2 = __hne2(h2, h2_1);
  // CHECK: sycl::ext::intel::math::hneu2(h2, h2);
  __hneu2(h2, h2);
  // CHECK: h2_2 = sycl::ext::intel::math::hadd2(h2, h2_1);
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2(h2, h2_1);
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2(h2, h2_1);
  h2_2 = __hsub2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hneg2(h2_1);
  h2_2 = __hneg2(h2_1);
}

int main() { return 0; }
