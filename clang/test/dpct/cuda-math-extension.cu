// RUN: dpct --format-range=none --use-dpcpp-extensions=intel_device_math -out-root %T/cuda-math-extension %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuda-math-extension/cuda-math-extension.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hip/hip_fp16.h"

using namespace std;

__global__ void kernelFuncHalf() {
  __half h, h_1, h_2;
  bool b;

  // Half Arithmetic Functions

  // CHECK: h_2 = sycl::ext::intel::math::hadd_sat(h, h_1);
  h_2 = __hadd_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hfma_sat(h, h_1, h_2);
  h_2 = __hfma_sat(h, h_1, h_2);
  // CHECK: h_2 = sycl::ext::intel::math::hmul_sat(h, h_1);
  h_2 = __hmul_sat(h, h_1);
  // CHECK: h_2 = sycl::ext::intel::math::hsub_sat(h, h_1);
  h_2 = __hsub_sat(h, h_1);

  // Half Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hequ(h, h_1);
  b = __hequ(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgeu(h, h_1);
  b = __hgeu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hgtu(h, h_1);
  b = __hgtu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hleu(h, h_1);
  b = __hleu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hltu(h, h_1);
  b = __hltu(h, h_1);
  // CHECK: b = sycl::ext::intel::math::hneu(h, h_1);
  b = __hneu(h, h_1);
}

__global__ void kernelFuncHalf2() {
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = sycl::ext::intel::math::hadd2_sat(h2, h2_1);
  h2_2 = __hadd2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hfma2_sat(h2, h2_1, h2_2);
  h2_2 = __hfma2_sat(h2, h2_1, h2_2);
  // CHECK: h2_2 = sycl::ext::intel::math::hmul2_sat(h2, h2_1);
  h2_2 = __hmul2_sat(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hsub2_sat(h2, h2_1);
  h2_2 = __hsub2_sat(h2, h2_1);

  // Half2 Comparison Functions

  // CHECK: b = sycl::ext::intel::math::hbeq2(h2, h2_1);
  b = __hbeq2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbequ2(h2, h2_1);
  b = __hbequ2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbge2(h2, h2_1);
  b = __hbge2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgeu2(h2, h2_1);
  b = __hbgeu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgt2(h2, h2_1);
  b = __hbgt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbgtu2(h2, h2_1);
  b = __hbgtu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hble2(h2, h2_1);
  b = __hble2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbleu2(h2, h2_1);
  b = __hbleu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hblt2(h2, h2_1);
  b = __hblt2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbltu2(h2, h2_1);
  b = __hbltu2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbne2(h2, h2_1);
  b = __hbne2(h2, h2_1);
  // CHECK: b = sycl::ext::intel::math::hbneu2(h2, h2_1);
  b = __hbneu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::heq2(h2, h2_1);
  h2_2 = __heq2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hequ2(h2, h2_1);
  h2_2 = __hequ2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hge2(h2, h2_1);
  h2_2 = __hge2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgeu2(h2, h2_1);
  h2_2 = __hgeu2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgt2(h2, h2_1);
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: h2_2 = sycl::ext::intel::math::hgtu2(h2, h2_1);
  h2_2 = __hgtu2(h2, h2_1);
}

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::ext::intel::math::erfcinv(d0);
  d2 = erfcinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::erfinv(d0);
  d2 = erfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorm(d0);
  d2 = normcdf(d0);
  // CHECK: d2 = sycl::ext::intel::math::cdfnorminv(d0);
  d2 = normcdfinv(d0);
  // CHECK: d2 = sycl::ext::intel::math::norm(i, (const double *)&d0);
  d2 = norm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::norm((int)d1, (const double *)&d0);
  d2 = norm(d1, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm(i, (const double *)&d0);
  d2 = rnorm(i, &d0);
  // CHECK: d2 = sycl::ext::intel::math::rnorm((int)d1, (const double *)&d0);
  d2 = rnorm(d1, &d0);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::ext::intel::math::erfcinv(f0);
  f2 = erfcinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfcinv((float)i);
  f2 = erfcinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::erfinv(f0);
  f2 = erfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::erfinv((float)i);
  f2 = erfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm(f0);
  f2 = normcdff(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorm((float)i);
  f2 = normcdff(i);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv(f0);
  f2 = normcdfinvf(f0);
  // CHECK: f2 = sycl::ext::intel::math::cdfnorminv((float)i);
  f2 = normcdfinvf(i);
  // CHECK: f2 = sycl::ext::intel::math::norm(i, (const float *)&f0);
  f2 = normf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::norm((int)f1, (const float *)&f0);
  f2 = normf(f1, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm(i, (const float *)&f0);
  f2 = rnormf(i, &f0);
  // CHECK: f2 = sycl::ext::intel::math::rnorm((int)f1, (const float *)&f0);
  f2 = rnormf(f1, &f0);
}

int main() { return 0; }
