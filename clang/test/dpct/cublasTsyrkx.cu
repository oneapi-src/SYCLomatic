// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasTsyrkx %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasTsyrkx/cublasTsyrkx.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublasTsyrkx/cublasTsyrkx.dp.cpp -o %T/cublasTsyrkx/cublasTsyrkx.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

//CHECK: #define macro_a (oneapi::mkl::transpose)1
#define macro_a (hipblasOperation_t)1

//CHECK: #define macro_b (oneapi::mkl::uplo)1
#define macro_b (hipblasFillMode_t)1

hipblasFillMode_t foo(){
  return HIPBLAS_FILL_MODE_LOWER;
}

hipblasOperation_t bar(){
  return HIPBLAS_OP_T;
}

int main() {
  int n = 275;
  int k = 275;
  int lda = 1;
  int ldb = 1;
  int ldc = 1;

  float alpha_s = 1;
  float beta_s = 1;

  double alpha_d = 1;
  double beta_d = 1;

  hipblasHandle_t handle;
  hipblasStatus_t status;

  float* A_s=0;
  float* B_s=0;
  float* C_s=0;

  double* A_d=0;
  double* B_d=0;
  double* C_d=0;

  int trans0 = 0;
  int trans1 = 1;
  int fill0 = 0;
  int fill1 = 1;


  //CHECK: status = DPCT_CHECK_ERROR(dpct::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc));
  //CHECK-NEXT: dpct::syrk(*handle, fill1 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);
  status = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);
  hipblasSsyrkx(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)trans1, n, k, &alpha_s, A_s, lda, B_s, ldb, &beta_s, C_s, ldc);

  //CHECK: status = DPCT_CHECK_ERROR(dpct::syrk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc));
  //CHECK-NEXT: dpct::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  status = hipblasDsyrkx(handle, (hipblasFillMode_t)0, (hipblasOperation_t)0, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, (hipblasFillMode_t)1, (hipblasOperation_t)1, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  //CHECK: dpct::syrk(*handle, foo(), oneapi::mkl::transpose::trans, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, foo(), macro_a, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  //CHECK: dpct::syrk(*handle, oneapi::mkl::uplo::upper, bar(), n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, macro_b, bar(), n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  //CHECK: dpct::syrk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::trans, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_T, n, k, &alpha_d, A_d, lda, B_d, ldb, &beta_d, C_d, ldc);

  return 0;
}

