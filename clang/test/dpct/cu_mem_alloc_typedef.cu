// UNSUPPORTED: system-linux
// RUN: dpct --format-range=none -out-root %T/cu_mem_alloc_typedef %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cu_mem_alloc_typedef/cu_mem_alloc_typedef.dp.cpp
#include <cstdint>
#include <hip/hip_runtime.h>

typedef uint64_t hipDeviceptr_t;

// CHECK: void foo(dpct::device_ptr ptr) {
void foo(hipDeviceptr_t ptr) {
  // CHECK: ptr = (dpct::device_ptr)sycl::malloc_device(1024, dpct::get_in_order_queue());
  hipMalloc(&ptr, 1024);
}
