
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/launch-kernel-cooperative-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/launch-kernel-cooperative-usm/launch-kernel-cooperative-usm.dp.cpp

// CHECK: void template_device(T *d, T *s) {
template<class T>
__device__ void template_device(T *d) {
  __shared__ T s[16];
}

// CHECK: void template_kernel(T *d, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT: uint8_t *dpct_local, T *s) {
template<class T>
__global__ void template_kernel(T *d) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ T es[];
  template_device(d);
}

// CHECK: void kernel(int *d, dpct::image_accessor_ext<int, 1> tex, const sycl::nd_item<3> &item_ct1) {
__global__ void kernel(int *d, hipTextureObject_t tex) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  tex1D(d + gtid, tex, gtid);
}

int main() {
  int *d_data;
  hipMalloc(&d_data, sizeof(int));

  int *d_data21;
  hipMalloc(&d_data21, sizeof(int) * 32);
  hipTextureObject_t tex;
  hipResourceDesc res;
  hipTextureDesc texDesc;
  res.resType = hipResourceTypeLinear;
  res.res.linear.devPtr = d_data21;
  res.res.linear.desc.f = hipChannelFormatKindSigned;
  res.res.linear.desc.x = sizeof(int)*8; // bits per channel
  res.res.linear.sizeInBytes = sizeof(int)*8;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.addressMode[2] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  hipCreateTextureObject(&tex, &res, &texDesc, NULL);

  void *args[2] = { &d_data, &tex };

  // CHECK: q_ct1.submit(
  // CHECK-NEXT:  [&](sycl::handler &cgh) {
  // CHECK-NEXT:    auto tex_acc = static_cast<dpct::image_wrapper<int, 1> *>(*(dpct::image_wrapper_base_p *)args[1])->get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:    auto tex_smpl = (*(dpct::image_wrapper_base_p *)args[1])->get_sampler();
  // CHECK-EMPTY:
  // CHECK-NEXT:    auto d_ct0 = *(int **)args[0];
  // CHECK-EMPTY:
  // CHECK-NEXT:    cgh.parallel_for(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 16) * sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        kernel(d_ct0, dpct::image_accessor_ext<int, 1>(tex_smpl, tex_acc), item_ct1);
  // CHECK-NEXT:      });
  // CHECK-NEXT:  });
  hipLaunchCooperativeKernel(reinterpret_cast<const void*>((void *)&kernel), dim3(16), dim3(16), args, 0, 0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // CHECK: stream->submit(
  // CHECK-NEXT:  [&](sycl::handler &cgh) {
  // CHECK-NEXT:    sycl::local_accessor<uint8_t, 1> dpct_local_acc_ct1(sycl::range<1>(32), cgh);
  // CHECK-NEXT:    sycl::local_accessor<int, 1> s_acc_ct1(sycl::range<1>(16), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:    auto d_ct0 = *(int **)args[0];
  // CHECK-EMPTY:
  // CHECK-NEXT:    cgh.parallel_for(
  // CHECK-NEXT:      sycl::nd_range<3>(sycl::range<3>(1, 1, 16) * sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16)),
  // CHECK-NEXT:      [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:        template_kernel<int>(d_ct0, item_ct1, dpct_local_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get(), s_acc_ct1.template get_multi_ptr<sycl::access::decorated::no>().get());
  // CHECK-NEXT:      });
  // CHECK-NEXT:  });
  hipLaunchCooperativeKernel(reinterpret_cast<const void*>((const void *)&template_kernel<int>), dim3(16), dim3(16), args, 32, stream);

  void *kernel_func = (void *)&kernel;

  // CHECK: /*
  // CHECK-NEXT: DPCT1123:{{[0-9]+}}: The kernel function pointer cannot be used in the device code. You need to call the kernel function with the correct argument(s) directly. According to the kernel function definition, adjusting the dimension of the sycl::nd_item may also be required.
  // CHECK-NEXT: */
  // CHECK-NEXT: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 16) * sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16)), 
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:     kernel_func();
  // CHECK-NEXT:   });
  hipLaunchCooperativeKernel(reinterpret_cast<const void*>(kernel_func), dim3(16), dim3(16), args, 0, 0);

  hipStreamDestroy(stream);
  hipDestroyTextureObject(tex);
  hipFree(d_data21);
  hipFree(d_data);
}

