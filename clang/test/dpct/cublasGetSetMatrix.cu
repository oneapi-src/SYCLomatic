// RUN: dpct --format-range=none --usm-level=none -extra-arg-before=-std=c++14 -out-root %T/cublasGetSetMatrix %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasGetSetMatrix/cublasGetSetMatrix.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublasGetSetMatrix/cublasGetSetMatrix.dp.cpp -o %T/cublasGetSetMatrix/cublasGetSetMatrix.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

constexpr int foo(int i) {
  return i;
}

int main() {
  int rowsA = 100;
  int colsA = 100;
  int lda = 100;
  int ldb = 100;
  float *A = NULL;
  float *d_A = NULL;
  hipblasStatus_t status;
  // CHECK: dpct::queue_ptr stream;
  hipStream_t stream;

#define LDA_MARCO 100
  const int ConstLda = 100;
  // CHECK: status = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, LDA_MARCO, 100, colsA, sizeof(A[0])));
  status = hipblasSetMatrix(100, colsA, sizeof(A[0]), A, LDA_MARCO, d_A, 100);

  // CHECK: dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, ConstLda, 100, colsA, sizeof(A[0]));
  hipblasSetMatrix(100, colsA, sizeof(A[0]), A, ConstLda, d_A, 100);

  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter(s) lda and/or ldb could not be evaluated, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, ldb, lda, rowsA, colsA, sizeof(A[0]));
  hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, lda, d_A, ldb);

#define LDB_MARCO 99
  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter 100 does not equal to parameter LDB_MARCO, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, LDB_MARCO, 100, rowsA, colsA, sizeof(A[0]));
  hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, 100, d_A, LDB_MARCO);

  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter rowsA could not be evaluated and may be smaller than parameter 100, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, rowsA, colsA, sizeof(A[0]));
  hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, 100, d_A, 100);

  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter 99 is smaller than parameter 100, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 99, colsA, sizeof(A[0]));
  hipblasSetMatrix(99, colsA, sizeof(A[0]), A, 100, d_A, 100);

  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter 99 is smaller than parameter 100, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 99, colsA, sizeof(A[0])));
  status = hipblasSetMatrix(99, colsA, sizeof(A[0]), A, 100, d_A, 100);

  const int ConstLdaNE = lda;
  const int ConstLdbNE = ldb;
  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasGetMatrix was migrated, but due to parameter(s) ConstLdaNE and/or ConstLdbNE could not be evaluated, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, ConstLdbNE, ConstLdaNE, rowsA, colsA, sizeof(A[0]));
  hipblasGetMatrix(rowsA, colsA, sizeof(A[0]), A, ConstLdaNE, d_A, ConstLdbNE);

  const int ConstLdaT = 100;
  const int ConstLdbT = 100;
  constexpr int ConstExprLda = 101;
  constexpr int ConstExprLdb = 101;
  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasSetMatrix was migrated, but due to parameter(s) foo(lda) and/or foo(ldb) could not be evaluated, the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, foo(ldb), foo(lda), rowsA, colsA, sizeof(A[0]));
  hipblasSetMatrix(rowsA, colsA, sizeof(A[0]), A, foo(lda), d_A, foo(ldb));

  // CHECK: dpct::matrix_mem_copy((void*)d_A, (void*)A, foo(ConstLdbT), foo(ConstLdaT), 100, colsA, sizeof(A[0]));
  hipblasSetMatrix(100, colsA, sizeof(A[0]), A, foo(ConstLdaT), d_A, foo(ConstLdbT));

  // CHECK: /*
  // CHECK-NEXT: DPCT1018:{{[0-9]+}}: The cublasGetMatrix was migrated, but due to parameter 100 is smaller than parameter foo(ConstExprLda), the generated code performance may be sub-optimal.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, ConstExprLdb, foo(ConstExprLda), 100, colsA, sizeof(A[0]));
  hipblasGetMatrix(100, colsA, sizeof(A[0]), A, foo(ConstExprLda), d_A, ConstExprLdb);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 100, colsA, sizeof(A[0]), dpct::automatic, *stream, true));
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 100, colsA, sizeof(A[0]), dpct::automatic, *stream, true);
  status = hipblasSetMatrixAsync(100, colsA, sizeof(A[0]), A, 100, d_A, 100, stream);
  hipblasSetMatrixAsync(100, colsA, sizeof(A[0]), A, 100, d_A, 100, stream);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 100, colsA, sizeof(A[0]), dpct::automatic, *stream, true));
  // CHECK-NEXT: dpct::matrix_mem_copy((void*)d_A, (void*)A, 100, 100, 100, colsA, sizeof(A[0]), dpct::automatic, *stream, true);
  status = hipblasGetMatrixAsync(100, colsA, sizeof(A[0]), A, 100, d_A, 100, stream);
  hipblasGetMatrixAsync(100, colsA, sizeof(A[0]), A, 100, d_A, 100, stream);

  return 0;
}

