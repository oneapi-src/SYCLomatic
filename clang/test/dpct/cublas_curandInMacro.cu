// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublas_curandInMacro %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas_curandInMacro/cublas_curandInMacro.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hiprand/hiprand.h>
#include <hip/hip_runtime.h>


#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

#define curandErrCheck(stat) { curandErrCheck_((stat), __FILE__, __LINE__); }
void curandErrCheck_(hiprandStatus_t stat, const char *file, int line) {
    if (stat != HIPRAND_STATUS_SUCCESS) {
        fprintf(stderr, "cuRand Error: %d %s %d\n", stat, file, line);
    }
}


int main() {
    hipblasHandle_t handle;
    int N = 275;
    float *d_A_S = 0;
    float *d_B_S = 0;
    float *d_C_S = 0;
    float alpha_S = 1.0f;
    float beta_S = 0.0f;
    int trans0 = 0;
    int trans1 = 1;
    int fill0 = 0;
    int side0 = 0;
    int diag0 = 0;
    int *result = 0;
    const float *x_S = 0;

    // CHECK: dpct::queue_ptr stream1;
    // CHECK-NEXT: stream1 = dpct::get_current_device().create_queue();
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cublasErrCheck((handle = stream1, 0));
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cublasErrCheck((stream1 = handle, 0));
    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cublasErrCheck(hipblasSetStream(handle, stream1));
    cublasErrCheck(hipblasGetStream(handle, &stream1));

    // CHECK: /*
    // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cublasErrCheck([&](){
    // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
    // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
    // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
    // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
    // CHECK-NEXT: return 0;
    // CHECK-NEXT: }());
    cublasErrCheck(hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));


    // CHECK: /*
    // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cublasErrCheck([&](){
    // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
    // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
    // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
    // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
    // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
    // CHECK-NEXT: } else {
    // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
    // CHECK-NEXT: }
    // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, N, x_S_buf_ct{{[0-9]+}}, N, res_temp_buf_ct{{[0-9]+}});
    // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
    // CHECK-NEXT: return 0;
    // CHECK-NEXT: }());
    cublasErrCheck(hipblasIsamax(handle, N, x_S, N, result));


    //CHECK: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: cublasErrCheck((dpct::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N), 0));
    cublasErrCheck(hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));


    //CHECK: /*
    //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT: */
    //CHECK-NEXT: cublasErrCheck((dpct::trmm(*handle, (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N), 0));
    cublasErrCheck(hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N));


    float2 *d_A_C = 0;
    float2 *d_B_C = 0;
    float2 *d_C_C = 0;
    float2 alpha_C;
    float2 beta_C;
    const float2 *x_C = 0;
    float **Aarray_S = 0;
    int *PivotArray = 0;
    int *infoArray = 0;
    int batchSize = 10;


    // CHECK: /*
    // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT:cublasErrCheck([&](){
    // CHECK-NEXT:auto d_A_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(d_A_C);
    // CHECK-NEXT:auto d_B_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(d_B_C);
    // CHECK-NEXT:auto d_C_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(d_C_C);
    // CHECK-NEXT:oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), d_A_C_buf_ct{{[0-9]+}}, N, d_B_C_buf_ct{{[0-9]+}}, N, std::complex<float>(beta_C.x(), beta_C.y()), d_C_C_buf_ct{{[0-9]+}}, N);
    // CHECK-NEXT:return 0;
    // CHECK-NEXT:}());
    cublasErrCheck(hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N));

    // CHECK: /*
    // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT:cublasErrCheck([&](){
    // CHECK-NEXT:auto x_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_C);
    // CHECK-NEXT:auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
    // CHECK-NEXT:sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
    // CHECK-NEXT:if (dpct::is_device_ptr(result)) {
    // CHECK-NEXT:  result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
    // CHECK-NEXT:} else {
    // CHECK-NEXT:  result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
    // CHECK-NEXT:}
    // CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(*handle, N, x_C_buf_ct{{[0-9]+}}, N, res_temp_buf_ct{{[0-9]+}});
    // CHECK-NEXT:result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
    // CHECK-NEXT:return 0;
    // CHECK-NEXT:}());
    cublasErrCheck(hipblasIcamax(handle, N, x_C, N, result));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cublasErrCheck((dpct::trmm(*handle, (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N), 0));
    cublasErrCheck(hipblasCtrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N));

    // CHECK: /*
    // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasSgetrfBatched. You may need to check the migrated code.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT:cublasErrCheck((dpct::getrf_batch_wrapper(*handle, N, Aarray_S, N, PivotArray, infoArray, batchSize), 0));
    cublasErrCheck(hipblasSgetrfBatched(handle, N, Aarray_S, N, PivotArray, infoArray, batchSize));



    float * __restrict__ d_data;
    //CHECK:dpct::rng::host_rng_ptr rng;
    //CHECK-NEXT:/*
    //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:*/
    //CHECK-NEXT:curandErrCheck((rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::philox4x32x10), 0));
    //CHECK-NEXT:/*
    //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:*/
    //CHECK-NEXT:curandErrCheck((rng->set_seed(1337ull), 0));
    //CHECK-NEXT:/*
    //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:*/
    //CHECK-NEXT:curandErrCheck((rng->generate_uniform(d_data, (100 + 1) * (200) * 4), 0));
    //CHECK-NEXT:/*
    //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    //CHECK-NEXT:*/
    //CHECK-NEXT:curandErrCheck((rng.reset(), 0));
    hiprandGenerator_t rng;
    curandErrCheck(hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_DEFAULT));
    curandErrCheck(hiprandSetPseudoRandomGeneratorSeed(rng, 1337ull));
    curandErrCheck(hiprandGenerateUniform(rng, d_data, (100 + 1) * (200) * 4));
    curandErrCheck(hiprandDestroyGenerator(rng));

}

