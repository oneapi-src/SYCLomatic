// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --use-experimental-features=bindless_images --format-range=none -out-root %T/cudaGraphics_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cudaGraphics_test/cudaGraphics_test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -DBUILD_TEST -fsycl %T/cudaGraphics_test/cudaGraphics_test.dp.cpp -o %T/cudaGraphics_test/cudaGraphics_test.dp.o %}

#include <hip/hip_runtime.h>
#ifdef _WIN32
#include <cuda_d3d11_interop.h>
#endif

int main() {
  // CHECK: dpct::experimental::interop_mem_wrapper_ptr resource;
  // CHECK-NEXT: dpct::experimental::interop_mem_wrapper_ptr *resources;
  // CHECK-NEXT: dpct::experimental::interop_mem_wrapper_ptr **resources_ptr;
  hipGraphicsResource_t resource;
  hipGraphicsResource_t *resources;
  hipGraphicsResource_t **resources_ptr;

  // CHECK: dpct::experimental::interop_mem_wrapper_ptr resources_arr[10];
  hipGraphicsResource_t resources_arr[10];

  // CHECK: dpct::experimental::interop_mem_wrapper_ptr resource1, *resources1, **resources_ptr1;
  hipGraphicsResource_t resource1, *resources1, **resources_ptr1;

  hipMipmappedArray_t mipmappedArray, *mipmappedArray_ptr;
  hipArray_t array, *array_ptr;

  // CHECK: int regFlags = 0;
  // CHECK-NEXT: int regFlags1 = 0;
  // CHECK-NEXT: int regFlags2 = 0;
  // CHECK-NEXT: int regFlags3 = 0;
  // CHECK-NEXT: int regFlags4 = 0;
  hipGraphicsRegisterFlags regFlags = hipGraphicsRegisterFlagsNone;
  hipGraphicsRegisterFlags regFlags1 = hipGraphicsRegisterFlagsReadOnly;
  hipGraphicsRegisterFlags regFlags2 = hipGraphicsRegisterFlagsWriteDiscard;
  hipGraphicsRegisterFlags regFlags3 = hipGraphicsRegisterFlagsSurfaceLoadStore;
  hipGraphicsRegisterFlags regFlags4 = hipGraphicsRegisterFlagsTextureGather;

#ifdef _WIN32
  ID3D11Resource *pD3DResource, *pD3DResource1;

  // CHECK-WINDOWS: resource = new dpct::experimental::interop_mem_wrapper(pD3DResource, 0);
  cudaGraphicsD3D11RegisterResource(&resource, pD3DResource, cudaGraphicsRegisterFlagsNone);

  // CHECK-WINDOWS: resource1 = new dpct::experimental::interop_mem_wrapper(pD3DResource1, regFlags1);
  cudaGraphicsD3D11RegisterResource(&resource1, pD3DResource1, regFlags1);
#endif

  resources_arr[0] = resource;
  resources_arr[1] = resource1;

  // CHECK: int mapFlags = 0;
  // CHECK-NEXT: int mapFlags1 = 0;
  // CHECK-NEXT: int mapFlags2 = 0;
  cudaGraphicsMapFlags mapFlags = cudaGraphicsMapFlagsNone;
  cudaGraphicsMapFlags mapFlags1 = cudaGraphicsMapFlagsReadOnly;
  cudaGraphicsMapFlags mapFlags2 = cudaGraphicsMapFlagsWriteDiscard;

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaGraphicsResourceSetMapFlags was removed because this functionality is deprecated in DX12 and hence is not supported in SYCL.
  // CHECK-NEXT: */
  cudaGraphicsResourceSetMapFlags(resource, cudaGraphicsMapFlagsNone);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cudaGraphicsResourceSetMapFlags was removed because this functionality is deprecated in DX12 and hence is not supported in SYCL.
  // CHECK-NEXT: */
  cudaGraphicsResourceSetMapFlags(resource1, mapFlags1);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // CHECK: dpct::experimental::map_resources(2, resources_arr, stream);
  hipGraphicsMapResources(2, resources_arr, stream);

  // CHECK: dpct::experimental::unmap_resources(2, resources_arr, stream);
  hipGraphicsUnmapResources(2, resources_arr, stream);

  // CHECK: dpct::experimental::map_resources(1, &resource);
  hipGraphicsMapResources(1, &resource);

  // CHECK: mipmappedArray = resource->get_mapped_mipmapped_array();
  cudaGraphicsResourceGetMappedMipmappedArray(&mipmappedArray, resource);

  // CHECK: *mipmappedArray_ptr = resource->get_mapped_mipmapped_array();
  cudaGraphicsResourceGetMappedMipmappedArray(mipmappedArray_ptr, resource);

  void* devPtr;
  size_t size;
  // CHECK: dpct::experimental::get_mapped_pointer(&devPtr, &size, resource);
  hipGraphicsResourceGetMappedPointer(&devPtr, &size, resource);

  unsigned int arrayIndex, mipLevel;
  // CHECK: array = resource->get_sub_resource_mapped_array(arrayIndex, mipLevel);
  hipGraphicsSubResourceGetMappedArray(&array, resource, arrayIndex, mipLevel);

  // CHECK: *array_ptr = resource->get_sub_resource_mapped_array(arrayIndex, mipLevel);
  hipGraphicsSubResourceGetMappedArray(array_ptr, resource, arrayIndex, mipLevel);

  // CHECK: dpct::experimental::unmap_resources(1, &resource);
  hipGraphicsUnmapResources(1, &resource);

  // CHECK: delete(resource);
  hipGraphicsUnregisterResource(resource);

  // CHECK: delete(resource1);
  hipGraphicsUnregisterResource(resource1);

  return 0;
}
