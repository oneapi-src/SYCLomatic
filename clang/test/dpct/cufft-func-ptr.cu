// RUN: cat %s > %T/cufft-func-ptr.cu
// RUN: cd %T
// RUN: dpct -out-root %T/cufft-func-ptr cufft-func-ptr.cu --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-func-ptr/cufft-func-ptr.dp.cpp --match-full-lines cufft-func-ptr.cu

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//CHECK:static int (*pt2CufftExec)(std::shared_ptr<dpct::fft::fft_engine>,
//CHECK-NEXT:                           sycl::double2 *, double *) =
//CHECK-NEXT:    [](std::shared_ptr<dpct::fft::fft_engine> engine, sycl::double2 in,
//CHECK-NEXT:       double out) {
//CHECK-NEXT:      engine->compute<sycl::double2, double>(
//CHECK-NEXT:        in, out, dpct::fft::fft_direction::backward);
//CHECK-NEXT:      return 0;
//CHECK-NEXT:    };
static hipfftResult (*pt2CufftExec)(hipfftHandle, hipfftDoubleComplex *,
                                    double *) = &hipfftExecZ2D;

int main() {
//CHECK:  std::shared_ptr<dpct::fft::fft_engine> plan1;
//CHECK-NEXT:  plan1 = std::make_shared<dpct::fft::fft_engine>(
//CHECK-NEXT:      10, dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  pt2CufftExec(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  pt2CufftExec(plan1, idata, odata);
  return 0;
}

int foo1() {
//CHECK:  typedef int (*Func_t)(std::shared_ptr<dpct::fft::fft_engine>, sycl::double2 *,
//CHECK-NEXT:                        double *);
  typedef hipfftResult (*Func_t)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  static Func_t FuncPtr = [](std::shared_ptr<dpct::fft::fft_engine> engine,
//CHECK-NEXT:                             sycl::double2 in, double out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::fft_direction::backward);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  static Func_t FuncPtr  = &hipfftExecZ2D;

//CHECK:  std::shared_ptr<dpct::fft::fft_engine> plan1;
//CHECK-NEXT:  plan1 = std::make_shared<dpct::fft::fft_engine>(
//CHECK-NEXT:      10, dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr(plan1, idata, odata);
  return 0;
}

int foo2() {
//CHECK:  using Func_t = int (*)(std::shared_ptr<dpct::fft::fft_engine>,
//CHECK-NEXT:    sycl::double2 *, double *);
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  Func_t FuncPtr2 = [](std::shared_ptr<dpct::fft::fft_engine> engine,
//CHECK-NEXT:                       sycl::double2 in, double out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::fft_direction::backward);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  Func_t FuncPtr2  = &hipfftExecZ2D;

//CHECK:  std::shared_ptr<dpct::fft::fft_engine> plan1;
//CHECK-NEXT:  plan1 = std::make_shared<dpct::fft::fft_engine>(
//CHECK-NEXT:      10, dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr2(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr2(plan1, idata, odata);
  return 0;
}

int foo3() {
//CHECK:  using Func_t = int (*)(std::shared_ptr<dpct::fft::fft_engine>,
//CHECK-NEXT:                         sycl::double2 *, double *);
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  Func_t FuncPtr3;
//CHECK-NEXT:  FuncPtr3 = [](std::shared_ptr<dpct::fft::fft_engine> engine, sycl::double2 in,
//CHECK-NEXT:                double out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::fft_direction::backward);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  Func_t FuncPtr3;
  FuncPtr3 = &hipfftExecZ2D;

//CHECK:  std::shared_ptr<dpct::fft::fft_engine> plan1;
//CHECK-NEXT:  plan1 = std::make_shared<dpct::fft::fft_engine>(
//CHECK-NEXT:      10, dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr3(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr3(plan1, idata, odata);
  return 0;
}

int foo4() {
//CHECK:  int (*FuncPtr4)(std::shared_ptr<dpct::fft::fft_engine>, sycl::double2 *,
//CHECK-NEXT:                  double *);
  hipfftResult (*FuncPtr4)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  FuncPtr4 = [](std::shared_ptr<dpct::fft::fft_engine> engine, sycl::double2 in,
//CHECK-NEXT:                double out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::fft_direction::backward);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  FuncPtr4 = &hipfftExecZ2D;

//CHECK:  std::shared_ptr<dpct::fft::fft_engine> plan1;
//CHECK-NEXT:  plan1 = std::make_shared<dpct::fft::fft_engine>(
//CHECK-NEXT:      10, dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr4(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr4(plan1, idata, odata);
  return 0;
}