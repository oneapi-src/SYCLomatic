// RUN: cat %s > %T/cufft-func-ptr.cu
// RUN: cd %T
// RUN: dpct -out-root %T/cufft-func-ptr cufft-func-ptr.cu --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-func-ptr/cufft-func-ptr.dp.cpp --match-full-lines cufft-func-ptr.cu

#include <hip/hip_runtime.h>
#include <hipfft/hipfft.h>

//CHECK:static int (*pt2CufftExec)(dpct::fft::fft_engine_ptr, sycl::double2 *,
//CHECK-NEXT:                               double *) = [](dpct::fft::fft_engine_ptr engine,
//CHECK-NEXT:                                              sycl::double2 *in, double *out) {
//CHECK-NEXT:      engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                             dpct::fft::transform_direction::bwd);
//CHECK-NEXT:      return 0;
//CHECK-NEXT:    };
static hipfftResult (*pt2CufftExec)(hipfftHandle, hipfftDoubleComplex *,
                                    double *) = &hipfftExecZ2D;

int main() {
//CHECK:  dpct::fft::fft_engine_ptr plan1;
//CHECK-NEXT:  plan1 = dpct::fft::fft_engine::create(
//CHECK-NEXT:      &dpct::get_default_queue(), 10,
//CHECK-NEXT:      dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  pt2CufftExec(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  pt2CufftExec(plan1, idata, odata);
  return 0;
}

int foo1() {
//CHECK:  typedef int (*Func_t)(dpct::fft::fft_engine_ptr, sycl::double2 *, double *);
  typedef hipfftResult (*Func_t)(hipfftHandle, hipfftDoubleComplex *, double *);

//     CHECK:  static Func_t FuncPtr = [](dpct::fft::fft_engine_ptr engine,
//CHECK-NEXT:                             sycl::double2 *in, double *out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::transform_direction::bwd);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  static Func_t FuncPtr  = &hipfftExecZ2D;

//CHECK:  dpct::fft::fft_engine_ptr plan1;
//CHECK-NEXT:  plan1 = dpct::fft::fft_engine::create(
//CHECK-NEXT:      &dpct::get_default_queue(), 10,
//CHECK-NEXT:      dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr(plan1, idata, odata);
  return 0;
}

int foo2() {
//CHECK:  using Func_t = int (*)(dpct::fft::fft_engine_ptr, sycl::double2 *, double *);
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

//     CHECK:  Func_t FuncPtr2 = [](dpct::fft::fft_engine_ptr engine, sycl::double2 *in,
//CHECK-NEXT:                       double *out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::transform_direction::bwd);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  Func_t FuncPtr2  = &hipfftExecZ2D;

//CHECK:  dpct::fft::fft_engine_ptr plan1;
//CHECK-NEXT:  plan1 = dpct::fft::fft_engine::create(
//CHECK-NEXT:      &dpct::get_default_queue(), 10,
//CHECK-NEXT:      dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr2(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr2(plan1, idata, odata);
  return 0;
}

int foo3() {
//CHECK:  using Func_t = int (*)(dpct::fft::fft_engine_ptr, sycl::double2 *, double *);
  using Func_t = hipfftResult (*)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  Func_t FuncPtr3;
//CHECK-NEXT:  FuncPtr3 = [](dpct::fft::fft_engine_ptr engine, sycl::double2 *in,
//CHECK-NEXT:                double *out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::transform_direction::bwd);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  Func_t FuncPtr3;
  FuncPtr3 = &hipfftExecZ2D;

//CHECK:  dpct::fft::fft_engine_ptr plan1;
//CHECK-NEXT:  plan1 = dpct::fft::fft_engine::create(
//CHECK-NEXT:      &dpct::get_default_queue(), 10, 
//CHECK-NEXT:      dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr3(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr3(plan1, idata, odata);
  return 0;
}

int foo4() {
//CHECK:  int (*FuncPtr4)(dpct::fft::fft_engine_ptr, sycl::double2 *, double *);
  hipfftResult (*FuncPtr4)(hipfftHandle, hipfftDoubleComplex *, double *);

//CHECK:  FuncPtr4 = [](dpct::fft::fft_engine_ptr engine, sycl::double2 *in,
//CHECK-NEXT:                double *out) {
//CHECK-NEXT:    engine->compute<sycl::double2, double>(in, out,
//CHECK-NEXT:                                           dpct::fft::transform_direction::bwd);
//CHECK-NEXT:    return 0;
//CHECK-NEXT:  };
  FuncPtr4 = &hipfftExecZ2D;

//CHECK:  dpct::fft::fft_engine_ptr plan1;
//CHECK-NEXT:  plan1 = dpct::fft::fft_engine::create(
//CHECK-NEXT:      &dpct::get_default_queue(), 10,
//CHECK-NEXT:      dpct::fft::fft_type::complex_double_to_real_double, 1);
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2 *idata;
//CHECK-NEXT:  FuncPtr4(plan1, idata, odata);
//CHECK-NEXT:  return 0;
  hipfftHandle plan1;
  hipfftPlan1d(&plan1, 10, HIPFFT_Z2D, 1);
  double* odata;
  double2* idata;
  FuncPtr4(plan1, idata, odata);
  return 0;
}