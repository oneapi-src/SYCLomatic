// RUN: dpct --format-range=none -out-root %T/zero_length_array %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/zero_length_array/zero_length_array.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/zero_length_array/zero_length_array.dp.cpp -o %T/zero_length_array/zero_length_array.dp.o %}


#include <hip/hip_runtime.h>
#include <stdio.h>

struct S {
// CHECK: int abc;
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1102:{{[0-9]+}}: Zero-length arrays are not permitted in SYCL device code.
// CHECK-NEXT: */
// CHECK-NEXT: int arr[0];
  int abc;
  int arr[0];
};

__global__ void k(S* s) {
  int *arr0 = s->arr;
}

int main() {
  S* s;
  hipMalloc(&s, sizeof(S) + sizeof(int) * 4);
  k<<<1,1>>>(s);
  return 0;
}
