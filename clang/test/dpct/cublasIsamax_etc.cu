// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasIsamax_etc %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasIsamax_etc/cublasIsamax_etc.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int foo();

int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int k = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;
  const float *A_S = 0;
  const float *B_S = 0;
  float *C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  const double *A_D = 0;
  const double *B_D = 0;
  double *C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;

  const float *x_S = 0;
  const double *x_D = 0;
  const float *y_S = 0;
  const double *y_D = 0;
  int incx = 1;
  int incy = 1;
  int *result = 0;
  float *result_S = 0;
  double *result_D = 0;
  //level1
  //cublasI<t>amax
  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_S)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_S)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamax(handle, n, x_S, incx, result);
  hipblasIsamax(handle, n, x_S, incx, result);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_D)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_D)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIdamax(handle, n, x_D, incx, result);
  hipblasIdamax(handle, n, x_D, incx, result);

  //cublasI<t>amin
  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_S)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_S)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamin(handle, n, x_S, incx, result);
  hipblasIsamin(handle, n, x_S, incx, result);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_D)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::result_memory_t<std::int64_t, int> res(*handle, result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_D)), incx, res.get_memory(), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIdamin(handle, n, x_D, incx, result);
  hipblasIdamin(handle, n, x_D, incx, result);

  //cublas<t>asum
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::asum(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSasum(handle, n, x_S, incx, result_S);
  hipblasSasum(handle, n, x_S, incx, result_S);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::asum(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDasum(handle, n, x_D, incx, result_D);
  hipblasDasum(handle, n, x_D, incx, result_D);

  //cublas<t>axpy
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(*handle, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(*handle, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);
  hipblasSaxpy(handle, n, &alpha_S, x_S, incx, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(*handle, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(*handle, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, n, &alpha_D, x_D, incx, result_D, incy);

  //cublas<t>copy
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasScopy(handle, n, x_S, incx, result_S, incy);
  hipblasScopy(handle, n, x_S, incx, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDcopy(handle, n, x_D, incx, result_D, incy);
  hipblasDcopy(handle, n, x_D, incx, result_D, incy);

  //cublas<t>dot
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} =  sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dot(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} =  sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);
  hipblasSdot(handle, n, x_S, incx, y_S, incy, result_S);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} =  sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dot(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} =  sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);
  hipblasDdot(handle, n, x_D, incx, y_D, incy, result_D);

  //cublas<t>nrm2
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::nrm2(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_S)) {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_S_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_S, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(*handle, n, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSnrm2(handle, n, x_S, incx, result_S);
  hipblasSnrm2(handle, n, x_S, incx, result_S);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::nrm2(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_D)) {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_D_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_D, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(*handle, n, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDnrm2(handle, n, x_D, incx, result_D);
  hipblasDnrm2(handle, n, x_D, incx, result_D);

  float *x_f = 0;
  float *y_f = 0;
  double *x_d = 0;
  double *y_d = 0;
  //cublas<t>rot
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy, dpct::get_value(x_S, *handle), dpct::get_value(y_S, *handle)));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy, dpct::get_value(x_S, *handle), dpct::get_value(y_S, *handle));
  // CHECK-NEXT: }
  status = hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);
  hipblasSrot(handle, n, x_f, incx, y_f, incy, x_S, y_S);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy, dpct::get_value(x_D, *handle), dpct::get_value(y_D, *handle)));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy, dpct::get_value(x_D, *handle), dpct::get_value(y_D, *handle));
  // CHECK-NEXT: }
  status = hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);
  hipblasDrot(handle, n, x_d, incx, y_d, incy, x_D, y_D);

  //cublas<t>rotg
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_f)) {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(*handle, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_f)) {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSrotg(handle, x_f, y_f, x_f, y_f);
  hipblasSrotg(handle, x_f, y_f, x_f, y_f);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_d)) {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(*handle, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_d)) {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDrotg(handle, x_d, y_d, x_d, y_d);
  hipblasDrotg(handle, x_d, y_d, x_d, y_d);

  //cublas<t>rotm
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotm(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy, x_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotm(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy, x_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);
  hipblasSrotm(handle, n, x_f, incx, y_f, incy, x_S);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotm(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy, x_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotm(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy, x_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);
  hipblasDrotm(handle, n, x_d, incx, y_d, incy, x_D);

  //cublas<t>rotmg
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_f)) {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(5));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotmg(*handle, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, dpct::get_value(x_S, *handle), y_f_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_f)) {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(x_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(1));
  // CHECK-NEXT:   y_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(y_f, sycl::range<1>(5));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotmg(*handle, x_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, y_f_buf_ct{{[0-9]+}}, dpct::get_value(x_S, *handle), y_f_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);
  hipblasSrotmg(handle, x_f, y_f, y_f, x_S, y_f);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_d)) {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(5));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotmg(*handle, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, dpct::get_value(x_D, *handle), y_d_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_d)) {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(x_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(1));
  // CHECK-NEXT:   y_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(y_d, sycl::range<1>(5));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotmg(*handle, x_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, y_d_buf_ct{{[0-9]+}}, dpct::get_value(x_D, *handle), y_d_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);
  hipblasDrotmg(handle, x_d, y_d, y_d, x_D, y_d);

  //cublas<t>scal
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, alpha_S, x_f_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, alpha_S, x_f_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasSscal(handle, n, &alpha_S, x_f, incx);
  hipblasSscal(handle, n, &alpha_S, x_f, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, alpha_D, x_d_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, alpha_D, x_d_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasDscal(handle, n, &alpha_D, x_d, incx);
  hipblasDscal(handle, n, &alpha_D, x_d, incx);

  //cublas<t>swap
  // CHECK: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  // CHECK-NEXT: auto y_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(*handle, n, x_f_buf_ct{{[0-9]+}}, incx, y_f_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSswap(handle, n, x_f, incx, y_f, incy);
  hipblasSswap(handle, n, x_f, incx, y_f, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  // CHECK-NEXT: auto y_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(*handle, n, x_d_buf_ct{{[0-9]+}}, incx, y_d_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDswap(handle, n, x_d, incx, y_d, incy);
  hipblasDswap(handle, n, x_d, incx, y_d, incy);

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  //level2
  //cublas<t>gbmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, m, n, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(*handle, oneapi::mkl::transpose::nontrans, m, n, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSgbmv(handle, (hipblasOperation_t)trans0, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(*handle, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, m, n, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(*handle, oneapi::mkl::transpose::nontrans, m, n, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDgbmv(handle, (hipblasOperation_t)trans1, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgbmv(handle, HIPBLAS_OP_N, m, n, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>gemv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSgemv(handle, (hipblasOperation_t)trans2, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSgemv(handle, HIPBLAS_OP_N, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDgemv(handle, (hipblasOperation_t)0, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>ger
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::ger(*handle, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::ger(*handle, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSger(handle, m, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::ger(*handle, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::ger(*handle, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, m, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int fill0 = 0;
  int fill1 = 1;
  //cublas<t>sbmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::sbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::sbmv(*handle, oneapi::mkl::uplo::upper, m, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSsbmv(handle, (hipblasFillMode_t)fill0, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::sbmv(*handle, fill1==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::sbmv(*handle, oneapi::mkl::uplo::upper, m, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDsbmv(handle, (hipblasFillMode_t)fill1, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsbmv(handle, HIPBLAS_FILL_MODE_UPPER, m, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spmv(*handle, oneapi::mkl::uplo::lower, n, alpha_S, x_S_buf_ct{{[0-9]+}}, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spmv(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSspmv(handle, (hipblasFillMode_t)0, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);
  hipblasSspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, y_S, incx, &beta_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spmv(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spmv(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDspmv(handle, (hipblasFillMode_t)1, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);
  hipblasDspmv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>spr
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spr(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSspr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S);
  hipblasSspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spr(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDspr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D);
  hipblasDspr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D);

  //cublas<t>spr2
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spr2(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasSspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S);
  hipblasSspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::spr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::spr2(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDspr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D);
  hipblasDspr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D);

  //cublas<t>symv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, lda, y_S_buf_ct{{[0-9]+}}, incx, beta_S, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasSsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  hipblasSsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, lda, y_D_buf_ct{{[0-9]+}}, incx, beta_D, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDsymv(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  hipblasDsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);

  //cublas<t>syr
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, result_S_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasSsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, result_S, lda);
  hipblasSsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, result_S, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, result_D_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasDsyr(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, result_D, lda);
  hipblasDsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, result_D, lda);

  //cublas<t>syr2
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto y_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(*handle, oneapi::mkl::uplo::upper, n, alpha_S, x_S_buf_ct{{[0-9]+}}, incx, y_S_buf_ct{{[0-9]+}}, incy, result_S_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasSsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  hipblasSsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_S, x_S, incx, y_S, incy, result_S, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto y_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(y_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(*handle, oneapi::mkl::uplo::upper, n, alpha_D, x_D_buf_ct{{[0-9]+}}, incx, y_D_buf_ct{{[0-9]+}}, incy, result_D_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasDsyr2(handle, (hipblasFillMode_t)fill0, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, &alpha_D, x_D, incx, y_D, incy, result_D, lda);

  int diag0 = 0;
  int diag1 = 1;
  //cublas<t>tbmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, (oneapi::mkl::diag)diag0, n, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, n, x_S, lda, result_S, incy);
  hipblasStbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, (oneapi::mkl::diag)diag1, n, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, n, x_D, lda, result_D, incy);
  hipblasDtbmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tbsv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::nonunit, n, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0, n, n, x_S, lda, result_S, incy);
  hipblasStbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_S, lda, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::unit, n, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtbsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)1, n, n, x_D, lda, result_D, incy);
  hipblasDtbsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, n, x_D, lda, result_D, incy);

  //cublas<t>tpmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_S_buf_ct{{[0-9]+}}, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_S_buf_ct{{[0-9]+}}, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_D_buf_ct{{[0-9]+}}, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_D_buf_ct{{[0-9]+}}, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>tpsv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_S_buf_ct{{[0-9]+}}, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_S_buf_ct{{[0-9]+}}, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, result_S, incy);
  hipblasStpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_D_buf_ct{{[0-9]+}}, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_D_buf_ct{{[0-9]+}}, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, result_D, incy);
  hipblasDtpsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, result_D, incy);

  //cublas<t>trmv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrmv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //cublas<t>trsv
  // CHECK: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_S);
  // CHECK-NEXT: auto result_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_S_buf_ct{{[0-9]+}}, lda, result_S_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasStrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_S, lda, result_S, incy);
  hipblasStrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_S, lda, result_S, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_D);
  // CHECK-NEXT: auto result_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, n, x_D_buf_ct{{[0-9]+}}, lda, result_D_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasDtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_D, lda, result_D, incy);
  hipblasDtrsv(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, n, x_D, lda, result_D, incy);

  //level3
  int side0 = 0;
  int side1 = 1;
  // cublas<T>gemmStridedBatched
  // CHECK: {
  // CHECK-NEXT: auto A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::trans, n, n, n, alpha_S, A_S_buf_ct{{[0-9]+}}, n, 16, B_S_buf_ct{{[0-9]+}}, n, 16, beta_S, C_S_buf_ct{{[0-9]+}}, n, 16, 10));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT: auto C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, n, n, n, alpha_S, A_S_buf_ct{{[0-9]+}}, n, 16, B_S_buf_ct{{[0-9]+}}, n, 16, beta_S, C_S_buf_ct{{[0-9]+}}, n, 16, 10);
  // CHECK-NEXT: }
  status = hipblasSgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, n, n, n, &alpha_S, A_S, n, 16, B_S, n, 16, &beta_S, C_S, n, 16, 10);
  hipblasSgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, n, n, n, &alpha_S, A_S, n, 16, B_S, n, 16, &beta_S, C_S, n, 16, 10);

  // CHECK: {
  // CHECK-NEXT: auto A_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, n, n, n, alpha_D, A_D_buf_ct{{[0-9]+}}, n, 16, B_D_buf_ct{{[0-9]+}}, n, 16, beta_D, C_D_buf_ct{{[0-9]+}}, n, 16, 10));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT: auto C_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(C_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, n, n, n, alpha_D, A_D_buf_ct{{[0-9]+}}, n, 16, B_D_buf_ct{{[0-9]+}}, n, 16, beta_D, C_D_buf_ct{{[0-9]+}}, n, 16, 10);
  // CHECK-NEXT: }
  status = hipblasDgemmStridedBatched(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, n, n, n, &alpha_D, A_D, n, 16, B_D, n, 16, &beta_D, C_D, n, 16, 10);
  hipblasDgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, n, n, n, &alpha_D, A_D, n, 16, B_D, n, 16, &beta_D, C_D, n, 16, 10);

  __half alpha_H, beta_H;
  __half* A_H, *B_H, *C_H;
  // CHECK: {
  // CHECK-NEXT: auto A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(A_H);
  // CHECK-NEXT: auto B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(B_H);
  // CHECK-NEXT: auto C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(C_H);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, n, n, n, alpha_H, A_H_buf_ct{{[0-9]+}}, n, 16, B_H_buf_ct{{[0-9]+}}, n, 16, beta_H, C_H_buf_ct{{[0-9]+}}, n, 16, 10);
  // CHECK-NEXT: }
  hipblasHgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, n, n, n, &alpha_H, A_H, n, 16, B_H, n, 16, &beta_H, C_H, n, 16, 10);

  const float** A_S_array;
  const float** B_S_array;
  float** C_S_array;
  const double** A_D_array;
  const double** B_D_array;
  double** C_D_array;
  const __half** A_H_array;
  const __half** B_H_array;
  __half** C_H_array;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasSgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasSgemmBatched(handle, trans3, trans3, n, n, n, &alpha_S, A_S_array, n, B_S_array, n, &beta_S, C_S_array, n, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasSgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasSgemmBatched(handle, trans3, trans3, n, n, n, &alpha_S, A_S_array, n, B_S_array, n, &beta_S, C_S_array, n, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasDgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasDgemmBatched(handle, trans3, trans3, n, n, n, &alpha_D, A_D_array, n, B_D_array, n, &beta_D, C_D_array, n, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasDgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasDgemmBatched(handle, trans3, trans3, n, n, n, &alpha_D, A_D_array, n, B_D_array, n, &beta_D, C_D_array, n, 10);
  status = hipblasSgemmBatched(handle, trans3, trans3, n, n, n, &alpha_S, A_S_array, n, B_S_array, n, &beta_S, C_S_array, n, 10);
  hipblasSgemmBatched(handle, trans3, trans3, n, n, n, &alpha_S, A_S_array, n, B_S_array, n, &beta_S, C_S_array, n, 10);
  status = hipblasDgemmBatched(handle, trans3, trans3, n, n, n, &alpha_D, A_D_array, n, B_D_array, n, &beta_D, C_D_array, n, 10);
  hipblasDgemmBatched(handle, trans3, trans3, n, n, n, &alpha_D, A_D_array, n, B_D_array, n, &beta_D, C_D_array, n, 10);

  // cublas<T>symm
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(*handle, (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_S)), ldb, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(*handle, oneapi::mkl::side::right, oneapi::mkl::uplo::lower, m, n, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_S)), ldb, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc);
  status = hipblasSsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(*handle, (oneapi::mkl::side)side1, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_D)), ldb, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(*handle, oneapi::mkl::side::right, oneapi::mkl::uplo::lower, m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_D)), ldb, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  status = hipblasDsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsymm(handle, HIPBLAS_SIDE_RIGHT, HIPBLAS_FILL_MODE_LOWER, m, n, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>syrk
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc);
  status = hipblasSsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);
  hipblasSsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, &beta_S, C_S, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  status = hipblasDsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);
  hipblasDsyrk(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, &beta_D, C_D, ldc);

  // cublas<T>syr2k
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_S)), ldb, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_S)), ldb, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc);
  status = hipblasSsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);
  hipblasSsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_S, A_S, lda, B_S, ldb, &beta_S, C_S, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_D)), ldb, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_D)), ldb, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  status = hipblasDsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);
  hipblasDsyr2k(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, n, k, &alpha_D, A_D, lda, B_D, ldb, &beta_D, C_D, ldc);

  // cublas<T>trsm
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, m, n, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc);
  status = hipblasStrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_S, A_S, lda, C_S, ldc);
  hipblasStrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_S, A_S, lda, C_S, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::right, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  status = hipblasDtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, &alpha_D, A_D, lda, C_D, ldc);
  hipblasDtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, &alpha_D, A_D, lda, C_D, ldc);


  // CHECK:   oneapi::mkl::blas::column_major::trsm(*handle, (oneapi::mkl::side)foo(), foo() == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(foo()), (oneapi::mkl::diag)foo(), m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  hipblasDtrsm(handle, (hipblasSideMode_t)foo(), (hipblasFillMode_t)foo(), (hipblasOperation_t)foo(), (hipblasDiagType_t)foo(), m, n, &alpha_D, A_D, lda, C_D, ldc);

  // CHECK: oneapi::mkl::blas::column_major::omatadd(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_S)), lda, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_S)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_S)), ldc);
  hipblasSgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, &alpha_S, A_S, lda, &beta_S, B_S, ldb, C_S, ldc);

  // CHECK: oneapi::mkl::blas::column_major::omatadd(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_D)), lda, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_D)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_D)), ldc);
  hipblasDgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, &alpha_D, A_D, lda, &beta_D, B_D, ldb, C_D, ldc);
  return 0;
}

