
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -usm-level=none -out-root %T/launch-kernel %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/launch-kernel/launch-kernel.dp.cpp

// CHECK: inline void template_device(T *d, T *s) {
template<class T>
__device__ void template_device(T *d) {
  __shared__ T s[16];
}

// CHECK: __dpct_inline__ void template_kernel(T *d, const sycl::nd_item<3> &item_ct1,
// CHECK-NEXT: uint8_t *dpct_local, T *s) {
template<class T>
__global__ void template_kernel(T *d) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  extern __shared__ T es[];
  template_device(d);
}

// CHECK: __dpct_inline__ void kernel(int *d, dpct::image_accessor_ext<int, 1> tex, const sycl::nd_item<3> &item_ct1) {
__global__ void kernel(int *d, hipTextureObject_t tex) {
  int gtid = blockIdx.x * blockDim.x + threadIdx.x;
  tex1D(d + gtid, tex, gtid);
}

int main() {
  int *d_data;
  hipMalloc(&d_data, sizeof(int));

  int *d_data21;
  hipMalloc(&d_data21, sizeof(int) * 32);
  hipTextureObject_t tex;
  hipResourceDesc res;
  hipTextureDesc texDesc;
  res.resType = hipResourceTypeLinear;
  res.res.linear.devPtr = d_data21;
  res.res.linear.desc.f = hipChannelFormatKindSigned;
  res.res.linear.desc.x = sizeof(int)*8; // bits per channel
  res.res.linear.sizeInBytes = sizeof(int)*8;
  texDesc.addressMode[0] = hipAddressModeClamp;
  texDesc.addressMode[1] = hipAddressModeClamp;
  texDesc.addressMode[2] = hipAddressModeClamp;
  texDesc.filterMode = hipFilterModeLinear;
  hipCreateTextureObject(&tex, &res, &texDesc, NULL);

  void *args[2] = { &d_data, &tex };

  // CHECK: q_ct1.submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     dpct::access_wrapper<int *> d_acc_ct0(*(int **)args[0], cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     auto tex_acc = static_cast<dpct::image_wrapper<int, 1> *>(*(dpct::image_wrapper_base_p *)args[1])->get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     auto tex_smpl = (*(dpct::image_wrapper_base_p *)args[1])->get_sampler();
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 16) * sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         kernel(d_acc_ct0.get_raw_pointer(), dpct::image_accessor_ext<int, 1>(tex_smpl, tex_acc), item_ct1);
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  hipLaunchKernel(reinterpret_cast<const void*>((void *)&kernel), dim3(16), dim3(16), args, 0, 0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  // CHECK: stream->submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     sycl::local_accessor<uint8_t, 1> dpct_local_acc_ct1(sycl::range<1>(32), cgh);
  // CHECK-NEXT:     sycl::local_accessor<int, 1> s_acc_ct1(sycl::range<1>(16), cgh);
  // CHECK-NEXT:     dpct::access_wrapper<int *> d_acc_ct0(*(int **)args[0], cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 16) * sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         template_kernel<int>(d_acc_ct0.get_raw_pointer(), item_ct1, dpct_local_acc_ct1.get_pointer(), s_acc_ct1.get_pointer());
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  hipLaunchKernel(reinterpret_cast<const void*>((const void *)&template_kernel<int>), dim3(16), dim3(16), args, 32, stream);

  void *kernel_func = (void *)&kernel;
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cudaLaunchKernel is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaLaunchKernel(kernel_func, sycl::range<3>(1, 1, 16), sycl::range<3>(1, 1, 16), args, 0, &q_ct1);
  hipLaunchKernel(reinterpret_cast<const void*>(kernel_func), dim3(16), dim3(16), args, 0, 0);

  hipStreamDestroy(stream);
  hipDestroyTextureObject(tex);
  hipFree(d_data21);
  hipFree(d_data);
}

