// RUN: dpct --format-range=none -out-root %T/thrust_system_error %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/thrust_system_error/thrust_system_error.dp.cpp --match-full-lines %s

#include <hip/hip_runtime_api.h>
#include <thrust/system/cuda/error.h>
#include <thrust/system_error.h>
#include <string>

void cuda_safe_call(hipError_t error, const std::string& message = "")
{
  if(error) {
// CHECK: throw std::system_error(error, std::generic_category(), message);
    throw thrust::system_error(error, thrust::cuda_category(), message);
  }
}

int main() {
// CHECK: dpct::err0 e = 1;  
  hipError_t e = hipErrorInvalidValue;  
  cuda_safe_call(e);
  return 0;
} 