#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/sync_api_ndrange_barrier %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sync_api_ndrange_barrier/sync_api_ndrange_barrier.dp.cpp

// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include "hip/hip_cooperative_groups.h"
namespace cg = cooperative_groups;
using namespace cooperative_groups;

// CHECK: void kernel(const sycl::stream &stream_ct1) {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1087:{{[0-9]+}}: SYCL currently does not support cross group synchronization, you can specify "--use-experimental-features=nd_range_barrier" to use the dpct helper function nd_range_barrier to migrate this_grid().
// CHECK-NEXT:  */
// CHECK-NEXT:  cg::grid_group grid = cg::this_grid();
// CHECK-NEXT:  stream_ct1 << "kernel run!\n";
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1087:{{[0-9]+}}: SYCL currently does not support cross group synchronization, you can specify "--use-experimental-features=nd_range_barrier" to use the dpct helper function nd_range_barrier to migrate grid.sync().
// CHECK-NEXT:  */
// CHECK-NEXT:  grid.sync();
// CHECK-NEXT:}
__global__ void kernel() {
  cg::grid_group grid = cg::this_grid();
  printf("kernel run!\n");
  grid.sync();
}

int main() {
// CHECK:  dpct::get_default_queue().submit(
// CHECK-NEXT:    [&](sycl::handler &cgh) {
// CHECK-NEXT:      sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:
// CHECK-NEXT:      cgh.parallel_for(
// CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2), sycl::range<3>(1, 1, 2)), 
// CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:          kernel(stream_ct1);
// CHECK-NEXT:        });
// CHECK-NEXT:    });
  kernel<<<2, 2>>>();

  hipDeviceSynchronize();
  return 0;
}