// UNSUPPORTED: cuda-8.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.1, v9.2
// RUN: dpct --no-dpcpp-extensions=device_info -out-root %T/device004 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device004/device004.dp.cpp

#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  hipDeviceProp_t properties;
  hipGetDeviceProperties(&properties, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1090:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with pciDeviceID. It was not migrated. You need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: const int id = properties.pciDeviceID;
  const int id = properties.pciDeviceID;
  // CHECK: /*
  // CHECK-NEXT: DPCT1090:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with uuid. It was not migrated. You need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: const std::array<unsigned char, 16> uuid = properties.uuid;
  const hipUUID uuid = properties.uuid;
  // CHECK: /*
  // CHECK-NEXT: DPCT1090:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with pciDeviceID. It was not migrated. You need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: properties.pciDeviceID = id;
  properties.pciDeviceID = id;
  // CHECK: /*
  // CHECK-NEXT: DPCT1090:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with uuid. It was not migrated. You need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: properties.uuid = uuid;
  properties.uuid = uuid;
}

#define CHECK_INTERNAL(err)                                                    \
  { auto err_ = (err); }

#define CHECK(err) CHECK_INTERNAL(err)

void foo() {
  int dev = 1;
  hipDeviceProp_t p;
  // CHECK: CHECK(DPCT_CHECK_ERROR(dpct::get_device(dev).get_device_info(p)));
  CHECK(hipGetDeviceProperties(&p, dev));
}
