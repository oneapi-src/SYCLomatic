// RUN: dpct --format-range=none -in-root %S -out-root %T/explicit_namespace_sycl %S/explicit_namespace_sycl.cu --cuda-include-path="%cuda-path/include" --use-explicit-namespace=sycl --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/explicit_namespace_sycl/explicit_namespace_sycl.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/explicit_namespace_sycl/explicit_namespace_sycl.dp.cpp -o %T/explicit_namespace_sycl/explicit_namespace_sycl.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: using namespace dpct;
#include <cmath>
#include <hip/hip_runtime.h>


__device__ float4 fun() {
  float4 a, b, c;
// CHECK: sycl::fma(a.x(), b.x(), c.x());
  __fmaf_rn(a.x, b.x, c.x);
// CHECK: return sycl::float4(sycl::fma(a.x(), b.x(), c.x()), sycl::fma(a.y(), b.y(), c.y()), sycl::fma(a.z(), b.z(), c.z()), sycl::fma(a.w(), b.w(), c.w()));
  return make_float4(__fmaf_rd(a.x, b.x, c.x), __fmaf_rz(a.y, b.y, c.y), __fmaf_rn(a.z, b.z, c.z), __fmaf_rn(a.w, b.w, c.w));
}


__global__ void kernel() {

}

void foo() {
// CHECK:   get_in_order_queue().parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
// CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, ceil(2.3)), sycl::range<3>(1, 1, 1)),
// CHECK-NEXT:         [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {
// CHECK-NEXT:           kernel();
// CHECK-NEXT:         });
  kernel<<< ceil(2.3), 1 >>>();
}

int main() {

}