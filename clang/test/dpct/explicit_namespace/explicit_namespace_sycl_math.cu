// RUN: dpct --format-range=none -in-root %S -out-root %T/explicit_namespace_sycl_math %S/explicit_namespace_sycl_math.cu --cuda-include-path="%cuda-path/include" --use-explicit-namespace=sycl-math --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/explicit_namespace_sycl_math/explicit_namespace_sycl_math.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: using namespace dpct;
// CHECK-NEXT: using namespace sycl;
#include <cmath>
#include <hip/hip_runtime.h>


__device__ float4 fun() {
  float4 a, b, c;
  // CHECK: sycl::fma(a[0], b[0], c[0]);
  __fmaf_rn(a.x, b.x, c.x);
  // CHECK: return mfloat4(sycl::fma(a[0], b[0], c[0]), sycl::fma(a[1], b[1], c[1]), sycl::fma(a[2], b[2], c[2]), sycl::fma(a[3], b[3], c[3]));
  return make_float4(__fmaf_rd(a.x, b.x, c.x), __fmaf_rz(a.y, b.y, c.y), __fmaf_rn(a.z, b.z, c.z), __fmaf_rn(a.w, b.w, c.w));
}


__global__ void kernel() {

}

void foo() {
  // CHECK:   get_default_queue().parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         nd_range<3>(range<3>(1, 1, ceil(2.3)), range<3>(1, 1, 1)),
  // CHECK-NEXT:         [=](nd_item<3> item_{{[0-9a-z]+}}) {
  // CHECK-NEXT:           kernel();
  // CHECK-NEXT:         });
  kernel<<< ceil(2.3), 1 >>>();
}

int main() {

}