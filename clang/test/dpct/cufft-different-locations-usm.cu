// RUN: dpct --format-range=none -out-root %T/cufft-different-locations-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-different-locations-usm/cufft-different-locations-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
double* odata;
double2* idata;

#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line ) {
  if (err != HIPFFT_SUCCESS) {
    fprintf(stderr, "Cufft error in file '%s' in line %i : %s.\n",
            __FILE__, __LINE__, "error" );
  }
}

int main() {
  hipfftHandle plan1;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int res1 = (plan1->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0);
  hipfftResult res1 = hipfftMakePlanMany(plan1, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int res2 = (plan1->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0);
  hipfftResult res2 = hipfftExecZ2D(plan1, idata, odata);

  hipfftHandle plan2;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res1 = (plan2->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0);
  res1 = hipfftMakePlanMany(plan2, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:res2 = (plan2->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0);
  res2 = hipfftExecZ2D(plan2, idata, odata);

  hipfftHandle plan3;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR((plan3->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0));
  HANDLE_CUFFT_ERROR(hipfftMakePlanMany(plan3, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR((plan3->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0));
  HANDLE_CUFFT_ERROR(hipfftExecZ2D(plan3, idata, odata));

  hipfftHandle plan4;
  hipfftHandle plan5;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if((plan4->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0)) {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} else if ((plan5->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0)) {
  //CHECK-NEXT:}
  if(hipfftMakePlanMany(plan4, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  } else if (hipfftMakePlanMany(plan5, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if ((plan4->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0)) {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} else if((plan5->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0)) {
  //CHECK-NEXT:}
  if (hipfftExecZ2D(plan4, idata, odata)) {
  } else if(hipfftExecZ2D(plan5, idata, odata)) {
  }

  hipfftHandle plan6;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(int res = (plan6->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0)) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftMakePlanMany(plan6, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)) {
  }
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:if(int res = (plan6->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0)) {
  //CHECK-NEXT:}
  if(hipfftResult res = hipfftExecZ2D(plan6, idata, odata)) {
  }

  hipfftHandle plan7;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (plan7->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);;) {
  //CHECK-NEXT:}
  for (hipfftMakePlanMany(plan7, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);;) {
  }
  //CHECK:for (plan7->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd);;) {
  //CHECK-NEXT:}
  for (hipfftExecZ2D(plan7, idata, odata);;) {
  }

  hipfftHandle plan8;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (;(plan8->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0);) {
  //CHECK-NEXT:}
  for (;hipfftMakePlanMany(plan8, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);) {
  }

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:for (;(plan8->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0);) {
  //CHECK-NEXT:}
  for (;hipfftExecZ2D(plan8, idata, odata);) {
  }

  hipfftHandle plan9;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:while ((plan9->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0) != 0) {
  //CHECK-NEXT:}
  while (hipfftMakePlanMany(plan9, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size) != 0) {
  }

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:while ((plan9->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0) != 0) {
  //CHECK-NEXT:}
  while (hipfftExecZ2D(plan9, idata, odata) != 0) {
  }

  hipfftHandle plan10;
  //CHECK:do {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} while ((plan10->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0));
  do {
  } while (hipfftMakePlanMany(plan10, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size));
  //CHECK:do {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:} while ((plan10->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0));
  do {
  } while (hipfftExecZ2D(plan10, idata, odata));

  hipfftHandle plan11;
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:switch (int stat = (plan11->commit(&q_ct1, 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0)){
  //CHECK-NEXT:}
  switch (int stat = hipfftMakePlanMany(plan11, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size)){
  }

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:switch (int stat = (plan11->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0)){
  //CHECK-NEXT:}
  switch (int stat = hipfftExecZ2D(plan11, idata, odata)){
  }
  return 0;
}

hipfftResult foo1(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:return (plan->commit(&dpct::get_default_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size), 0);
  return hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo2(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:return (plan->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd), 0);
  return hipfftExecZ2D(plan, idata, odata);
}

hipfftResult foo3(hipfftHandle plan) {
  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported for GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(&dpct::get_default_queue(), 3, n, inembed, istride, idist, onembed, ostride, odist, dpct::fft::fft_type::complex_double_to_real_double, 12, work_size);
  hipfftMakePlanMany(plan, 3, n, inembed, istride, idist, onembed, ostride, odist, HIPFFT_Z2D, 12, work_size);
}

hipfftResult foo4(hipfftHandle plan) {
  //CHECK:plan->compute<sycl::double2, double>(idata, odata, dpct::fft::transform_direction::bwd);
  hipfftExecZ2D(plan, idata, odata);
}

