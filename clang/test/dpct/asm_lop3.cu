// RUN: dpct -out-root %T/asm_lop3 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm_lop3/asm_lop3.dp.cpp


#include <hip/hip_runtime.h>
#include <cstdint>

// a^b^c
static __device__ __forceinline__ uint32_t LOP3LUT_XOR(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d1;
  // CHECK: d1 = a ^ b ^ c;
  asm("lop3.b32 %0, %1, %2, %3, 0x96;" : "=r"(d1) : "r"(a), "r"(b), "r"(c));
  return d1;
}

// (a ^ (c & (b ^ a)))
static __device__ __forceinline__ uint32_t LOP3LUT_XORAND(uint32_t a, uint32_t b, uint32_t c) {
  uint32_t d2;
  // CHECK: d2 = (a ^ (c & (b ^ a)));
  asm("lop3.b32 %0, %1, %3, %2, 0xb8;" : "=r"(d2) : "r"(a), "r"(b), "r"(c));
  return d2;
}

// ((a & (b | b)) | (b & b))
static __device__ __forceinline__ uint32_t LOP3LUT_ANDOR(uint32_t a, uint32_t b) {
  uint32_t d3;
  // CHECK: d3 = ((a & (b | b)) | (b & b));
  asm("lop3.b32 %0, %1, %2, %2, 0xe8;" : "=r"(d3) : "r"(a), "r"(b));
  return d3;
}

#define B 3
__device__  int hard(int a) {
  int d4;
  // CHECK: d4 = (~((a + B) * (a + B)) & B & ~(3)) | (~((a + B) * (a + B)) & B & (3)) | (((a + B) * (a + B)) & ~B & ~(3));
  asm("lop3.b32 %0, %1 * %1, %2, 3, 0x1C;" : "=r"(d4) : "r"(a + B), "r"(B));
  return d4;
}
