// RUN: dpct --format-range=none --usm-level=none -out-root %T/queue_ctn_usm_none %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/queue_ctn_usm_none/queue_ctn_usm_none.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/queue_ctn_usm_none/queue_ctn_usm_none.dp.cpp -o %T/queue_ctn_usm_none/queue_ctn_usm_none.dp.o %}


#include "hip/hip_runtime.h"

void bar();
#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;
__constant__ float constData[123 * 4];

// CHECK: void foo1() {
// CHECK-NEXT: dpct::dpct_memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, dpct::device_to_host );
// CHECK-NEXT: dpct::dpct_memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, dpct::device_to_host );
// CHECK-NEXT: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
// CHECK-NEXT: dpct::dpct_memset(d_A, 23, size);
// CHECK-NEXT: dpct::dpct_memset(d_A, 23, size);
// CHECK-NEXT: bar();
// CHECK-NEXT: dpct::dpct_memset(d_A, 23, size);
// CHECK-NEXT: dpct::dpct_memset(d_A, 23, size);
// CHECK-NEXT: }
void foo1() {
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  bar();
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}

