// RUN: dpct --format-range=none --out-root %T/cusparse-usm-2 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm-2/cusparse-usm-2.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: oneapi::mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int foo(int aaaaa){
  //CHECK: oneapi::mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT: oneapi::mkl::index_base descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: int type0 = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetMatDiagType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetMatFillMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = (oneapi::mkl::index_base)aaaaa;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusparseGetMatDiagType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: diag0 = (oneapi::mkl::diag)0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusparseGetMatFillMode was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: fill0 = (oneapi::mkl::uplo)0;
  //CHECK-NEXT: base0 = descrA;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cusparseGetMatType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: type0 = 0;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

    //CHECK:int status;
  hipsparseStatus_t status;

 //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}


