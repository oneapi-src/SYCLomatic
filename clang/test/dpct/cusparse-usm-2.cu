// RUN: dpct --format-range=none --out-root %T/cusparse-usm-2 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm-2/cusparse-usm-2.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int foo(int aaaaa){
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> descr1 = 0, descr2 = 0;
  //CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descr3 = 0;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: dpct::sparse::matrix_info::matrix_type type0 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: descrA->set_diag((oneapi::mkl::diag)aaaaa);
  //CHECK-NEXT: descrA->set_uplo((oneapi::mkl::uplo)aaaaa);
  //CHECK-NEXT: descrA->set_index_base((oneapi::mkl::index_base)aaaaa);
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)aaaaa);
  //CHECK-NEXT: diag0 = descrA->get_diag();
  //CHECK-NEXT: fill0 = descrA->get_uplo();
  //CHECK-NEXT: base0 = descrA->get_index_base();
  //CHECK-NEXT: type0 = descrA->get_matrix_type();
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = std::make_shared<dpct::sparse::matrix_info>();
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)aaaaa);
  //CHECK-NEXT: descrA->set_index_base(oneapi::mkl::index_base::zero);
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);

  int version;
  //CHECK: dpct::mkl_get_version(dpct::version_field::major, &version);
  cusparseGetProperty(MAJOR_VERSION, &version);

  return 0;
}
