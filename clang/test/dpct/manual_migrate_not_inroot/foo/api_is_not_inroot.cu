// RUN: cd %T
// RUN: cat %S/../../nccl_test.h > %T/../nccl_test.h
// RUN: cat %S/../../cudnn_test.h > %T/../cudnn_test.h
// RUN: cat %s > %T/api_is_not_inroot.cu
// RUN: dpct --format-range=none ./api_is_not_inroot.cu --in-root=. --out-root=%T/out --cuda-include-path="%cuda-path/include" -- --cuda-host-only -I..
// RUN: FileCheck %s --match-full-lines --input-file %T/out/api_is_not_inroot.dp.cpp
// RUN: cd ..
// RUN: rm -rf ./*


//cudnn_test.h and nccl_test.h are not in inroot, so emit warnings.

//CHECK:#include <CL/sycl.hpp>
//CHECK-NEXT:#include <dpct/dpct.hpp>
//CHECK-NEXT:#include <cstdio>
//CHECK-NEXT:#include <cudnn_test.h>
#include <cstdio>
#include <cudnn_test.h>
#include <hip/hip_runtime.h>

int main() {

int a1 = cudnnAAA();
cudnnFooType b1;
cudnnFooEnum c1;
if(b1 == CUDNN_FOO_VAL){
}
cudnnCLASS d1;
cudnnTemplateCLASS<double> e1;
}
