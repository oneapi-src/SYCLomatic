// UNSUPPORTED: cuda-8.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3
// UNSUPPORTED: v8.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3
// RUN: dpct --usm-level=none --format-range=none -out-root %T/thrust-for-h2o4gpu-specific-case-noneusm %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/thrust-for-h2o4gpu-specific-case-noneusm/thrust-for-h2o4gpu-specific-case-noneusm.dp.cpp --match-full-lines %s

// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT:#include <oneapi/dpl/algorithm>
// CHECK-EMPTY:
// CHECK-NEXT:#define DPCT_USM_LEVEL_NONE
// CHECK-NEXT:#include <sycl/sycl.hpp>
// CHECK-NEXT:#include <dpct/dpct.hpp>
// CHECK-NEXT:#include <dpct/dpl_utils.hpp>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/device_allocator.h>
#include <thrust/device_malloc_allocator.h>
#include <thrust/iterator/iterator_traits.h>

// CHECK: /*
// CHECK-NEXT:DPCT1082:{{[0-9]+}}: Migration of thrust::device_malloc_allocator type is not supported.
// CHECK-NEXT:*/
// CHECK-NEXT:template <typename T> void foo_cpy(dpct::device_vector<T, thrust::device_malloc_allocator<T>> &Do, dpct::device_vector<T, thrust::device_malloc_allocator<T>> &Di) {
// CHECK-NEXT: return;
// CHECK-NEXT:}
template <typename T> void foo_cpy(thrust::device_vector<T, thrust::device_malloc_allocator<T>> &Do, thrust::device_vector<T, thrust::device_malloc_allocator<T>> &Di) {
 return;
}


void foo_1() {
// CHECK:  /*
// CHECK-NEXT:  DPCT1082:{{[0-9]+}}: Migration of thrust::device_malloc_allocator type is not supported.
// CHECK-NEXT:  */
// CHECK-NEXT:  dpct::device_vector<int, thrust::device_malloc_allocator<int>> **tt=NULL;
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1082:{{[0-9]+}}: Migration of thrust::device_malloc_allocator type is not supported.
// CHECK-NEXT:  */
// CHECK-NEXT:  dpct::device_vector<int, thrust::device_malloc_allocator<int>> *dd[10];
// CHECK-NEXT:  foo_cpy(*tt[0], *dd[0]);
  thrust::device_vector<int, thrust::device_malloc_allocator<int>> **tt=NULL;
  thrust::device_vector<int, thrust::device_malloc_allocator<int>> *dd[10];
  foo_cpy(*tt[0], *dd[0]);
}

// CHECK:/*
// CHECK-NEXT:DPCT1082:{{[0-9]+}}: Migration of thrust::device_malloc_allocator type is not supported.
// CHECK-NEXT:*/
namespace foo_struct { template <typename T, typename MemorySpace> struct default_memory_allocator : thrust::device_malloc_allocator<T>{};}
