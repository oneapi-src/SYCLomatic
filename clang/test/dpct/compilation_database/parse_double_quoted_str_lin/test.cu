// UNSUPPORTED: system-windows
// RUN: cd %T
// RUN: cat %S/compile_commands.json > %T/compile_commands.json
// RUN: cat %s > %T/test.cu
// RUN: dpct --format-range=none -in-root=%T  -out-root=%T/out -p ./ --format-range=none --cuda-include-path="%cuda-path/include" -gen-build-script
// RUN: FileCheck %s --match-full-lines --input-file %T/out/test.dp.cpp
// RUN: %if build_lit %{icpx -DNAMD="\"3.0b3\"" -c -fsycl %T/out/test.dp.cpp -o %T/out/test.dp.o %}
// RUN: cd %T/out
// RUN: %if build_lit %{make -f Makefile.dpct %}

// CHECK:  #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <iostream>
#include <hip/hip_runtime.h>
#include <iostream>
int main() {
  std::cout << NAMD;
  printf(NAMD);
  return 0;
}
