// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-12.0, cuda-12.1, cuda-12.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v12.0, v12.1, v12.2
// RUN: dpct --format-range=none --out-root %T/cusparse-type10 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cusparse-type10/cusparse-type10.dp.cpp --match-full-lines %s

#include <hipsparse.h>

int main() {
  //CHECK:int alg = 0;
  //CHECK-NEXT:alg = 1;
  cusparseAlgMode_t alg = CUSPARSE_ALG_NAIVE;
  alg = CUSPARSE_ALG_MERGE_PATH;

  return 0;
}
