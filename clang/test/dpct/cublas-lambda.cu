// RUN: cat %s > %T/cublas-lambda.cu
// RUN: cd %T
// RUN: dpct --no-cl-namespace-inline --usm-level=none -out-root %T/cublas-lambda cublas-lambda.cu --cuda-include-path="%cuda-path/include"  -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-lambda/cublas-lambda.dp.cpp --match-full-lines cublas-lambda.cu
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/cublas-lambda/cublas-lambda.dp.cpp -o %T/cublas-lambda/cublas-lambda.dp.o %}

#ifndef BUILD_TEST
// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <dpct/blas_utils.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

hipblasStatus_t status;
hipblasHandle_t handle;
int N = 275;
__half *d_A_H = 0;
__half *d_B_H = 0;
__half *d_C_H = 0;
__half alpha_H = 1.0f;
__half beta_H = 0.0f;

int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  // CHECK: handle = &dpct::get_out_of_order_queue();
  hipblasCreate(&handle);

  // CHECK: {
  // CHECK-NEXT:   auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT:   auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT:   auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT:   oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:       *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:       oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:       d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error
  // CHECK-NEXT: codes. 0 is used instead of an error code in an if statement. You may need to
  // CHECK-NEXT: rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (0) {
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the
  // CHECK-NEXT: lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: else if ([&]() {
  // CHECK-NEXT:            auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT:            auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT:            auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT:            oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:                *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:                oneapi::mkl::transpose::nontrans, N, N, N, alpha_H,
  // CHECK-NEXT:                d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT:            return 0;
  // CHECK-NEXT:          }()) {
  // CHECK-NEXT: }
  if (hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N)) {
  }
  else if (hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N)) {
  }


  // CHECK: {
  // CHECK-NEXT:   auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT:   auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT:   auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT:   oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:       *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:       oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:       d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error
  // CHECK-NEXT: codes. 0 is used instead of an error code in an if statement. You may need to
  // CHECK-NEXT: rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (int stat = 0) {
  // CHECK-NEXT: }
  if(int stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N)){
  }


  // CHECK: {
  // CHECK-NEXT:   auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT:   auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT:   auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT:   oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:       *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:       oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:       d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error
  // CHECK-NEXT: codes. 0 is used instead of an error code in a for statement. You may need to
  // CHECK-NEXT: rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: for (0;;) {
  // CHECK-NEXT: }
  for(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);;){
  }

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the
  // CHECK-NEXT: lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: for (; [&]() {
  // CHECK-NEXT: auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT: auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT: auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:     *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:     oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}},
  // CHECK-NEXT:     N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();) {
  // CHECK-NEXT: }
  for(;hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);){
  }

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the
  // CHECK-NEXT: lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: while ([&]() {
  // CHECK-NEXT: auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT: auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT: auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:     *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:     oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:     d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }() != 0) {
  // CHECK-NEXT: }
  while(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N)!=0){
  }



  // CHECK: do{
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the
  // CHECK-NEXT: lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: } while ([&]() {
  // CHECK-NEXT: auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT: auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT: auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:     *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:     oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:     d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }());
  do{
  }while(hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N));


  // CHECK: {
  // CHECK-NEXT:   auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
  // CHECK-NEXT:   auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
  // CHECK-NEXT:   auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
  // CHECK-NEXT:   oneapi::mkl::blas::column_major::gemm(
  // CHECK-NEXT:       *handle, oneapi::mkl::transpose::nontrans,
  // CHECK-NEXT:       oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
  // CHECK-NEXT:       d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error
  // CHECK-NEXT: codes. 0 is used instead of an error code in a switch statement. You may need
  // CHECK-NEXT: to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: switch (int stat = 0) {
  // CHECK-NEXT: }
  switch (int stat = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N)){
  }


  return 0;
}

// CHECK:int foo() try {
// CHECK-NEXT:  auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
// CHECK-NEXT:  auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
// CHECK-NEXT:  auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
// CHECK-NEXT:  oneapi::mkl::blas::column_major::gemm(
// CHECK-NEXT:      *handle, oneapi::mkl::transpose::nontrans,
// CHECK-NEXT:      oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
// CHECK-NEXT:      d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error
// CHECK-NEXT:  codes. 0 is used instead of an error code in a return statement. You may need
// CHECK-NEXT:  to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT:  return 0;
// CHECK-NEXT:}
int foo() {
  return hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
}

// CHECK:void foo2() {
// CHECK-NEXT:  auto d_A_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_A_H);
// CHECK-NEXT:  auto d_B_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_B_H);
// CHECK-NEXT:  auto d_C_H_buf_ct{{[0-9]+}} = dpct::get_buffer<cl::sycl::half>(d_C_H);
// CHECK-NEXT:  oneapi::mkl::blas::column_major::gemm(
// CHECK-NEXT:      *handle, oneapi::mkl::transpose::nontrans,
// CHECK-NEXT:      oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N,
// CHECK-NEXT:      d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
// CHECK-NEXT:}
void foo2() {
  hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
}
#endif
