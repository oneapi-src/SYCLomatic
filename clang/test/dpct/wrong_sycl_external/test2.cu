// RUN: echo
#include "hip/hip_runtime.h"

namespace {
// CHECK: inline void test_device() {}
__device__ void test_device() {}
}

template<typename T>
__global__ void test_global2(T a) {

    test_device();

}

int host_func() {
    int a;
    test_global2<<<1,1>>>(a);
}