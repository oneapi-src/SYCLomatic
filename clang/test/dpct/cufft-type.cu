#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/cufft-type %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-type/cufft-type.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>

size_t size;

int main() {
  //CHECK:float fftreal;
  //CHECK-NEXT:double fftdreal;
  //CHECK-NEXT:sycl::float2 fftcomplex;
  //CHECK-NEXT:sycl::double2 fftdcomplex;
  //CHECK-NEXT:sycl::float2 ccomplex;
  //CHECK-NEXT:sycl::double2 cdcomplex;
  //CHECK-NEXT:size = sizeof(float);
  //CHECK-NEXT:size = sizeof(double);
  //CHECK-NEXT:size = sizeof(sycl::float2);
  //CHECK-NEXT:size = sizeof(sycl::double2);
  //CHECK-NEXT:size = sizeof(sycl::float2);
  //CHECK-NEXT:size = sizeof(sycl::double2);
  hipfftReal fftreal;
  hipfftDoubleReal fftdreal;
  hipfftComplex fftcomplex;
  hipfftDoubleComplex fftdcomplex;
  hipComplex ccomplex;
  hipDoubleComplex cdcomplex;
  size = sizeof(hipfftReal);
  size = sizeof(hipfftDoubleReal);
  size = sizeof(hipfftComplex);
  size = sizeof(hipfftDoubleComplex);
  size = sizeof(hipComplex);
  size = sizeof(hipDoubleComplex);

  //CHECK:int forward = dpct::fft::fft_direction::forward;
  //CHECK-NEXT:int inverse = dpct::fft::fft_direction::backward;
  int forward = HIPFFT_FORWARD;
  int inverse = HIPFFT_BACKWARD;

  //CHECK:dpct::fft::fft_type fftt_t;
  //CHECK-NEXT:dpct::fft::fft_type fftt;
  //CHECK-NEXT:size = sizeof(dpct::fft::fft_type);
  //CHECK-NEXT:size = sizeof(dpct::fft::fft_type);
  //CHECK-NEXT:fftt = dpct::fft::fft_type::real_float_to_complex_float;
  //CHECK-NEXT:fftt = dpct::fft::fft_type::complex_float_to_real_float;
  //CHECK-NEXT:fftt = dpct::fft::fft_type::complex_float_to_complex_float;
  //CHECK-NEXT:fftt = dpct::fft::fft_type::real_double_to_complex_double;
  //CHECK-NEXT:fftt = dpct::fft::fft_type::complex_double_to_real_double;
  //CHECK-NEXT:fftt = dpct::fft::fft_type::complex_double_to_complex_double;
  hipfftType_t fftt_t;
  hipfftType fftt;
  size = sizeof(hipfftType_t);
  size = sizeof(hipfftType);
  fftt = HIPFFT_R2C;
  fftt = HIPFFT_C2R;
  fftt = HIPFFT_C2C;
  fftt = HIPFFT_D2Z;
  fftt = HIPFFT_Z2D;
  fftt = HIPFFT_Z2Z;

  //CHECK:dpct::fft::fft_engine* ffth;
  //CHECK-NEXT:size = sizeof(dpct::fft::fft_engine*);
  hipfftHandle ffth;
  size = sizeof(hipfftHandle);

  //CHECK:int fftr_t;
  //CHECK-NEXT:int fftr;
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:size = sizeof(int);
  //CHECK-NEXT:fftr = 0;
  //CHECK-NEXT:fftr = 1;
  //CHECK-NEXT:fftr = 2;
  //CHECK-NEXT:fftr = 3;
  //CHECK-NEXT:fftr = 4;
  //CHECK-NEXT:fftr = 5;
  //CHECK-NEXT:fftr = 6;
  //CHECK-NEXT:fftr = 7;
  //CHECK-NEXT:fftr = 8;
  //CHECK-NEXT:fftr = 9;
  //CHECK-NEXT:fftr = 10;
  //CHECK-NEXT:fftr = 11;
  //CHECK-NEXT:fftr = 12;
  //CHECK-NEXT:fftr = 13;
  //CHECK-NEXT:fftr = 14;
  //CHECK-NEXT:fftr = 15;
  //CHECK-NEXT:fftr = 16;
  hipfftResult_t fftr_t;
  hipfftResult fftr;
  size = sizeof(hipfftResult_t);
  size = sizeof(hipfftResult);
  fftr = HIPFFT_SUCCESS;
  fftr = HIPFFT_INVALID_PLAN;
  fftr = HIPFFT_ALLOC_FAILED;
  fftr = HIPFFT_INVALID_TYPE;
  fftr = HIPFFT_INVALID_VALUE;
  fftr = HIPFFT_INTERNAL_ERROR;
  fftr = HIPFFT_EXEC_FAILED;
  fftr = HIPFFT_SETUP_FAILED;
  fftr = HIPFFT_INVALID_SIZE;
  fftr = HIPFFT_UNALIGNED_DATA;
  fftr = HIPFFT_INCOMPLETE_PARAMETER_LIST;
  fftr = HIPFFT_INVALID_DEVICE;
  fftr = HIPFFT_PARSE_ERROR;
  fftr = HIPFFT_NO_WORKSPACE;
  fftr = HIPFFT_NOT_IMPLEMENTED;
  fftr = CUFFT_LICENSE_ERROR;
  fftr = HIPFFT_NOT_SUPPORTED;

  return 0;
}


//CHECK:template<
//CHECK-NEXT:typename A = float,
//CHECK-NEXT:typename B = double,
//CHECK-NEXT:typename C = sycl::float2,
//CHECK-NEXT:typename D = sycl::double2,
//CHECK-NEXT:typename E = sycl::float2,
//CHECK-NEXT:typename F = sycl::double2,
//CHECK-NEXT:typename G = dpct::fft::fft_type,
//CHECK-NEXT:typename H = dpct::fft::fft_type,
//CHECK-NEXT:typename I = dpct::fft::fft_engine*,
//CHECK-NEXT:typename J = int,
//CHECK-NEXT:typename K = int>
//CHECK-NEXT:void foo1(
//CHECK-NEXT:float a,
//CHECK-NEXT:double b,
//CHECK-NEXT:sycl::float2 c,
//CHECK-NEXT:sycl::double2 d,
//CHECK-NEXT:sycl::float2 e,
//CHECK-NEXT:sycl::double2 f,
//CHECK-NEXT:dpct::fft::fft_type g,
//CHECK-NEXT:dpct::fft::fft_type h,
//CHECK-NEXT:dpct::fft::fft_engine* i,
//CHECK-NEXT:int j,
//CHECK-NEXT:int k
//CHECK-NEXT:){}
template<
typename A = hipfftReal,
typename B = hipfftDoubleReal,
typename C = hipfftComplex,
typename D = hipfftDoubleComplex,
typename E = hipComplex,
typename F = hipDoubleComplex,
typename G = hipfftType_t,
typename H = hipfftType,
typename I = hipfftHandle,
typename J = hipfftResult_t,
typename K = hipfftResult>
void foo1(
hipfftReal a,
hipfftDoubleReal b,
hipfftComplex c,
hipfftDoubleComplex d,
hipComplex e,
hipDoubleComplex f,
hipfftType_t g,
hipfftType h,
hipfftHandle i,
hipfftResult_t j,
hipfftResult k
){}


//CHECK:template<
//CHECK-NEXT:dpct::fft::fft_type A1 = dpct::fft::fft_type::real_float_to_complex_float,
//CHECK-NEXT:dpct::fft::fft_type A2 = dpct::fft::fft_type::complex_float_to_real_float,
//CHECK-NEXT:dpct::fft::fft_type A3 = dpct::fft::fft_type::complex_float_to_complex_float,
//CHECK-NEXT:dpct::fft::fft_type A4 = dpct::fft::fft_type::real_double_to_complex_double,
//CHECK-NEXT:dpct::fft::fft_type A5 = dpct::fft::fft_type::complex_double_to_real_double,
//CHECK-NEXT:dpct::fft::fft_type A6 = dpct::fft::fft_type::complex_double_to_complex_double,
//CHECK-NEXT:int B1 = 0,
//CHECK-NEXT:int B2 = 1,
//CHECK-NEXT:int B3 = 2,
//CHECK-NEXT:int B4 = 3,
//CHECK-NEXT:int B5 = 4,
//CHECK-NEXT:int B6 = 5,
//CHECK-NEXT:int B7 = 6,
//CHECK-NEXT:int B8 = 7,
//CHECK-NEXT:int B9 = 8,
//CHECK-NEXT:int B10 = 9,
//CHECK-NEXT:int B11 = 10,
//CHECK-NEXT:int B12 = 11,
//CHECK-NEXT:int B13 = 12,
//CHECK-NEXT:int B14 = 13,
//CHECK-NEXT:int B15 = 14,
//CHECK-NEXT:int B16 = 15,
//CHECK-NEXT:int B17 = 16>
//CHECK-NEXT:void foo2(
//CHECK-NEXT:dpct::fft::fft_type a1 = dpct::fft::fft_type::real_float_to_complex_float,
//CHECK-NEXT:dpct::fft::fft_type a2 = dpct::fft::fft_type::complex_float_to_real_float,
//CHECK-NEXT:dpct::fft::fft_type a3 = dpct::fft::fft_type::complex_float_to_complex_float,
//CHECK-NEXT:dpct::fft::fft_type a4 = dpct::fft::fft_type::real_double_to_complex_double,
//CHECK-NEXT:dpct::fft::fft_type a5 = dpct::fft::fft_type::complex_double_to_real_double,
//CHECK-NEXT:dpct::fft::fft_type a6 = dpct::fft::fft_type::complex_double_to_complex_double,
//CHECK-NEXT:int b1 = 0,
//CHECK-NEXT:int b2 = 1,
//CHECK-NEXT:int b3 = 2,
//CHECK-NEXT:int b4 = 3,
//CHECK-NEXT:int b5 = 4,
//CHECK-NEXT:int b6 = 5,
//CHECK-NEXT:int b7 = 6,
//CHECK-NEXT:int b8 = 7,
//CHECK-NEXT:int b9 = 8,
//CHECK-NEXT:int b10 = 9,
//CHECK-NEXT:int b11 = 10,
//CHECK-NEXT:int b12 = 11,
//CHECK-NEXT:int b13 = 12,
//CHECK-NEXT:int b14 = 13,
//CHECK-NEXT:int b15 = 14,
//CHECK-NEXT:int b16 = 15,
//CHECK-NEXT:int b17 = 16
//CHECK-NEXT:){}
template<
hipfftType A1 = HIPFFT_R2C,
hipfftType A2 = HIPFFT_C2R,
hipfftType A3 = HIPFFT_C2C,
hipfftType A4 = HIPFFT_D2Z,
hipfftType A5 = HIPFFT_Z2D,
hipfftType A6 = HIPFFT_Z2Z,
hipfftResult B1 = HIPFFT_SUCCESS,
hipfftResult B2 = HIPFFT_INVALID_PLAN,
hipfftResult B3 = HIPFFT_ALLOC_FAILED,
hipfftResult B4 = HIPFFT_INVALID_TYPE,
hipfftResult B5 = HIPFFT_INVALID_VALUE,
hipfftResult B6 = HIPFFT_INTERNAL_ERROR,
hipfftResult B7 = HIPFFT_EXEC_FAILED,
hipfftResult B8 = HIPFFT_SETUP_FAILED,
hipfftResult B9 = HIPFFT_INVALID_SIZE,
hipfftResult B10 = HIPFFT_UNALIGNED_DATA,
hipfftResult B11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult B12 = HIPFFT_INVALID_DEVICE,
hipfftResult B13 = HIPFFT_PARSE_ERROR,
hipfftResult B14 = HIPFFT_NO_WORKSPACE,
hipfftResult B15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult B16 = CUFFT_LICENSE_ERROR,
hipfftResult B17 = HIPFFT_NOT_SUPPORTED>
void foo2(
hipfftType a1 = HIPFFT_R2C,
hipfftType a2 = HIPFFT_C2R,
hipfftType a3 = HIPFFT_C2C,
hipfftType a4 = HIPFFT_D2Z,
hipfftType a5 = HIPFFT_Z2D,
hipfftType a6 = HIPFFT_Z2Z,
hipfftResult b1 = HIPFFT_SUCCESS,
hipfftResult b2 = HIPFFT_INVALID_PLAN,
hipfftResult b3 = HIPFFT_ALLOC_FAILED,
hipfftResult b4 = HIPFFT_INVALID_TYPE,
hipfftResult b5 = HIPFFT_INVALID_VALUE,
hipfftResult b6 = HIPFFT_INTERNAL_ERROR,
hipfftResult b7 = HIPFFT_EXEC_FAILED,
hipfftResult b8 = HIPFFT_SETUP_FAILED,
hipfftResult b9 = HIPFFT_INVALID_SIZE,
hipfftResult b10 = HIPFFT_UNALIGNED_DATA,
hipfftResult b11 = HIPFFT_INCOMPLETE_PARAMETER_LIST,
hipfftResult b12 = HIPFFT_INVALID_DEVICE,
hipfftResult b13 = HIPFFT_PARSE_ERROR,
hipfftResult b14 = HIPFFT_NO_WORKSPACE,
hipfftResult b15 = HIPFFT_NOT_IMPLEMENTED,
hipfftResult b16 = CUFFT_LICENSE_ERROR,
hipfftResult b17 = HIPFFT_NOT_SUPPORTED
){}


//CHECK:template<typename T>
//CHECK-NEXT:float foo3(){}
template<typename T>
hipfftReal foo3(){}

//CHECK:template<typename T>
//CHECK-NEXT:double foo4(){}
template<typename T>
hipfftDoubleReal foo4(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::float2 foo5(){}
template<typename T>
hipfftComplex foo5(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::double2 foo6(){}
template<typename T>
hipfftDoubleComplex foo6(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::float2 foo7(){}
template<typename T>
hipComplex foo7(){}

//CHECK:template<typename T>
//CHECK-NEXT:sycl::double2 foo8(){}
template<typename T>
hipDoubleComplex foo8(){}

//CHECK:template<typename T>
//CHECK-NEXT:dpct::fft::fft_type foo9(){}
template<typename T>
hipfftType_t foo9(){}

//CHECK:template<typename T>
//CHECK-NEXT:dpct::fft::fft_type foo10(){}
template<typename T>
hipfftType foo10(){}

//CHECK:template<typename T>
//CHECK-NEXT:dpct::fft::fft_engine* foo11(){}
template<typename T>
hipfftHandle foo11(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo12(){}
template<typename T>
hipfftResult_t foo12(){}

//CHECK:template<typename T>
//CHECK-NEXT:int foo13(){}
template<typename T>
hipfftResult foo13(){}

//     CHECK:void bar1(dpct::fft::fft_engine* const &aaa) {}
//CHECK-NEXT:void bar2(dpct::fft::fft_engine* const &aaa) {}
//CHECK-NEXT:void bar3(dpct::fft::fft_engine* const aaa) {}
//CHECK-NEXT:void bar4(dpct::fft::fft_engine* const aaa) {}
//CHECK-NEXT:void bar5(dpct::fft::fft_engine* const *aaa) {}
//CHECK-NEXT:void bar6(dpct::fft::fft_engine* const *aaa) {}
void bar1(hipfftHandle const &aaa) {}
void bar2(const hipfftHandle &aaa) {}
void bar3(hipfftHandle const aaa) {}
void bar4(const hipfftHandle aaa) {}
void bar5(hipfftHandle const *aaa) {}
void bar6(const hipfftHandle *aaa) {}
