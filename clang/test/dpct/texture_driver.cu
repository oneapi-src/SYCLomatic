// RUN: dpct --format-range=none --usm-level=none -out-root %T/texture_driver %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14 -fno-delayed-template-parsing
// RUN: FileCheck --input-file %T/texture_driver/texture_driver.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  int err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

void func(int i) {}

template <typename T>
void funcT(T t) {}

int main() {

  // CHECK: size_t halfDesc_x_ct1, halfDesc_y_ct1;
  // CHECK-NEXT: unsigned halfDesc_channel_num_ct1;
  // CHECK-NEXT: sycl::image_channel_type halfDesc_channel_type_ct1;
  // CHECK-NEXT: halfDesc_y_ct1 = 32;
  // CHECK-NEXT: halfDesc_x_ct1 = 64;
  // CHECK-NEXT: halfDesc_channel_type_ct1 = sycl::image_channel_type::fp16;
  // CHECK-NEXT: halfDesc_channel_num_ct1 = 1;
  HIP_ARRAY_DESCRIPTOR halfDesc;
  halfDesc.Height = 32;
  halfDesc.Width = 64;
  halfDesc.Format = HIP_AD_FORMAT_HALF;
  halfDesc.NumChannels = 1;

  // CHECK: size_t float4Desc_x_ct1, float4Desc_y_ct1;
  // CHECK-NEXT: unsigned float4Desc_channel_num_ct1;
  // CHECK-NEXT: sycl::image_channel_type float4Desc_channel_type_ct1;
  // CHECK-NEXT: float4Desc_x_ct1 = 64;
  // CHECK-NEXT: float4Desc_channel_type_ct1 = sycl::image_channel_type::fp32;
  // CHECK-NEXT: float4Desc_channel_num_ct1 = 4;
  // CHECK-NEXT: float4Desc_y_ct1 = 32;
  HIP_ARRAY_DESCRIPTOR float4Desc;
  float4Desc.Width = 64;
  float4Desc.Format = HIP_AD_FORMAT_FLOAT;
  float4Desc.NumChannels = 4;
  float4Desc.Height = 32;

  // CHECK: dpct::image_matrix **a_ptr = new dpct::image_matrix_p;
  // CHECK-NEXT: dpct::image_matrix_p a42;
  // CHECK-NEXT: *a_ptr = new dpct::image_matrix(halfDesc_channel_type_ct1, halfDesc_channel_num_ct1, halfDesc_x_ct1, halfDesc_y_ct1);
  // CHECK-NEXT: a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1);
  // CHECK-NEXT: delete (*a_ptr);
  // CHECK-NEXT: delete a42;
  // CHECK-NEXT: delete a_ptr;
  hipArray **a_ptr = new hipArray_t;
  hipArray_t a42;
  hipArrayCreate(a_ptr, &halfDesc);
  hipArrayCreate(&a42, &float4Desc);
  hipArrayDestroy(*a_ptr);
  hipArrayDestroy(a42);
  delete a_ptr;

  // Test IsAssigned
  {
    int errorCode;

    // CHECK: errorCode = (a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1), 0);
    errorCode = hipArrayCreate(&a42, &float4Desc);
    // CHECK: errorCode = (delete a42, 0);
    errorCode = hipArrayDestroy(a42);


    // CHECK: cudaCheck((a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1), 0));
    cudaCheck(hipArrayCreate(&a42, &float4Desc));
    // CHECK: cudaCheck((delete a42, 0));
    cudaCheck(hipArrayDestroy(a42));


    // CHECK: func((a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1), 0));
    func(hipArrayCreate(&a42, &float4Desc));
    // CHECK: func((delete a42, 0));
    func(hipArrayDestroy(a42));


    // CHECK: funcT((a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1), 0));
    funcT(hipArrayCreate(&a42, &float4Desc));
    // CHECK: funcT((delete a42, 0));
    funcT(hipArrayDestroy(a42));
  }
}

void create_array_fail() {
  hipArray_t a;
  unsigned i;
  // CHECK: CUDA_ARRAY_DESCRIPTOR d[20], *p;
  HIP_ARRAY_DESCRIPTOR d[20], *p;
  p = &d[5];

  // CHECK: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'd' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, d);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'p' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, p);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'p + i' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, p + i);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter '&d[i]' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, &d[i]);
  hipArrayCreate(&a, d);
  hipArrayCreate(&a, p);
  hipArrayCreate(&a, p + i);
  hipArrayCreate(&a, &d[i]);
}

void test_texref() {
  // CHECK: sycl::addressing_mode addr_mode;
  // CHECK-NEXT: sycl::filtering_mode filter_mode;
  // CHECK-NEXT: sycl::image_channel_type format;
  // CHECK-NEXT: dpct::image_matrix_p arr;
  // CHECK-NEXT: dpct::image_wrapper_base_p tex;
  // CHECK-NEXT: int err_code;
  HIPaddress_mode addr_mode;
  HIPfilter_mode filter_mode;
  hipArray_Format format;
  hipArray_t arr;
  hipTexRef tex;
  hipError_t err_code;
  int flags, chn_num;

  // CHECK: tex->set_channel_type(format);
  // CHECK-NEXT: tex->set_channel_num(4);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = (tex->set_channel_type(sycl::image_channel_type::fp32), tex->set_channel_num(chn_num), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaCheck((tex->set_channel_type(format), tex->set_channel_num(4), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: func((tex->set_channel_type(format), tex->set_channel_num(4), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: funcT((tex->set_channel_type(format), tex->set_channel_num(4), 0));
  hipTexRefSetFormat(tex, format, 4);
  err_code = hipTexRefSetFormat(tex, HIP_AD_FORMAT_FLOAT, chn_num);
  cudaCheck(hipTexRefSetFormat(tex, format, 4));
  func(hipTexRefSetFormat(tex,format,4));
  funcT(hipTexRefSetFormat(tex,format,4));
  
  // CHECK: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: tex->set_coordinate_normalization_mode(flags & 0x02);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = (tex->set(sycl::coordinate_normalization_mode::normalized), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaCheck((tex->set(sycl::coordinate_normalization_mode::normalized), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: func((tex->set(sycl::coordinate_normalization_mode::normalized), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: funcT((tex->set(sycl::coordinate_normalization_mode::unnormalized), 0));
  hipTexRefSetFlags(tex, flags);
  err_code = hipTexRefSetFlags(tex, HIP_TRSF_NORMALIZED_COORDINATES);
  cudaCheck(hipTexRefSetFlags(tex,  HIP_TRSF_NORMALIZED_COORDINATES | HIP_TRSF_READ_AS_INTEGER));
  func(hipTexRefSetFlags(tex,3));
  funcT(hipTexRefSetFlags(tex,1));

  // CHECK: tex->set(addr_mode);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = (tex->set(sycl::addressing_mode::clamp_to_edge), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaCheck((tex->set(addr_mode), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: func((tex->set(addr_mode), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: funcT((tex->set(addr_mode), 0));
  hipTexRefSetAddressMode(tex, 0, addr_mode);
  err_code = hipTexRefSetAddressMode(tex, 1, HIP_TR_ADDRESS_MODE_CLAMP);
  cudaCheck(hipTexRefSetAddressMode(tex, 2, addr_mode));
  func(hipTexRefSetAddressMode(tex,0,addr_mode));
  funcT(hipTexRefSetAddressMode(tex,0,addr_mode));

  // CHECK: tex->set(filter_mode);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = (tex->set(sycl::filtering_mode::linear), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaCheck((tex->set(filter_mode), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: func((tex->set(filter_mode), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: funcT((tex->set(filter_mode), 0));
  hipTexRefSetFilterMode(tex, filter_mode);
  err_code = hipTexRefSetFilterMode(tex, HIP_TR_FILTER_MODE_LINEAR);
  cudaCheck(hipTexRefSetFilterMode(tex, filter_mode));
  func(hipTexRefSetFilterMode(tex,filter_mode));
  funcT(hipTexRefSetFilterMode(tex,filter_mode));

  // CHECK: tex->attach(dpct::image_data(arr));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = (tex->attach(dpct::image_data(arr)), 0);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaCheck((tex->attach(dpct::image_data(arr)), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: func((tex->attach(dpct::image_data(arr)), 0));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: funcT((tex->attach(dpct::image_data(arr)), 0));
  hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT);
  err_code = hipTexRefSetArray(tex, arr, 0x01);
  cudaCheck(hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT));
  func(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));
  funcT(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));

  // CHECK: char * dptr;
  // CHECK-Next: size_t s, b;
  // CHECK-Next: tex->attach(dptr, b);
  // CHECK-Next: size_t desc_x_ct1, desc_y_ct1;
  // CHECK-Next: unsigned desc_channel_num_ct1;
  // CHECK-Next: sycl::image_channel_type desc_channel_type_ct1;
  // CHECK-Next: tex->attach(dptr, desc_x_ct1, desc_y_ct1, b);
  hipDeviceptr_t dptr;
  size_t s, b;
  hipTexRefSetAddress(&s, tex, dptr, b);
  HIP_ARRAY_DESCRIPTOR desc;
  hipTexRefSetAddress2D(tex, &desc, dptr, b);
}