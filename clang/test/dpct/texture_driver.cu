// RUN: dpct --format-range=none --usm-level=none -out-root %T/texture_driver %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture_driver/texture_driver.dp.cpp --match-full-lines %s

#include "hip/hip_runtime.h"
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  int err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

void func(int i) {}

template <typename T>
void funcT(T t) {}

int main() {

  // CHECK: size_t halfDesc_x_ct1, halfDesc_y_ct1;
  // CHECK-NEXT: unsigned halfDesc_channel_num_ct1;
  // CHECK-NEXT: sycl::image_channel_type halfDesc_channel_type_ct1;
  // CHECK-NEXT: halfDesc_y_ct1 = 32;
  // CHECK-NEXT: halfDesc_x_ct1 = 64;
  // CHECK-NEXT: halfDesc_channel_type_ct1 = sycl::image_channel_type::fp16;
  // CHECK-NEXT: halfDesc_channel_num_ct1 = 1;
  HIP_ARRAY_DESCRIPTOR halfDesc;
  halfDesc.Height = 32;
  halfDesc.Width = 64;
  halfDesc.Format = HIP_AD_FORMAT_HALF;
  halfDesc.NumChannels = 1;

  // CHECK: size_t float4Desc_x_ct1, float4Desc_y_ct1;
  // CHECK-NEXT: unsigned float4Desc_channel_num_ct1;
  // CHECK-NEXT: sycl::image_channel_type float4Desc_channel_type_ct1;
  // CHECK-NEXT: float4Desc_x_ct1 = 64;
  // CHECK-NEXT: float4Desc_channel_type_ct1 = sycl::image_channel_type::fp32;
  // CHECK-NEXT: float4Desc_channel_num_ct1 = 4;
  // CHECK-NEXT: float4Desc_y_ct1 = 32;
  HIP_ARRAY_DESCRIPTOR float4Desc;
  float4Desc.Width = 64;
  float4Desc.Format = HIP_AD_FORMAT_FLOAT;
  float4Desc.NumChannels = 4;
  float4Desc.Height = 32;

  // CHECK: dpct::image_matrix **a_ptr = new dpct::image_matrix_p;
  // CHECK-NEXT: dpct::image_matrix_p a42;
  // CHECK-NEXT: *a_ptr = new dpct::image_matrix(halfDesc_channel_type_ct1, halfDesc_channel_num_ct1, halfDesc_x_ct1, halfDesc_y_ct1);
  // CHECK-NEXT: a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1);
  // CHECK-NEXT: delete (*a_ptr);
  // CHECK-NEXT: delete a42;
  // CHECK-NEXT: delete a_ptr;
  hipArray **a_ptr = new hipArray_t;
  hipArray_t a42;
  hipArrayCreate(a_ptr, &halfDesc);
  hipArrayCreate(&a42, &float4Desc);
  hipArrayDestroy(*a_ptr);
  hipArrayDestroy(a42);
  delete a_ptr;

  // Test IsAssigned
  {
    int errorCode;

    // CHECK: errorCode = DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1));
    errorCode = hipArrayCreate(&a42, &float4Desc);
    // CHECK: errorCode = DPCT_CHECK_ERROR(delete a42);
    errorCode = hipArrayDestroy(a42);


    // CHECK: cudaCheck(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1)));
    cudaCheck(hipArrayCreate(&a42, &float4Desc));
    // CHECK: cudaCheck(DPCT_CHECK_ERROR(delete a42));
    cudaCheck(hipArrayDestroy(a42));


    // CHECK: func(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1)));
    func(hipArrayCreate(&a42, &float4Desc));
    // CHECK: func(DPCT_CHECK_ERROR(delete a42));
    func(hipArrayDestroy(a42));


    // CHECK: funcT(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(float4Desc_channel_type_ct1, float4Desc_channel_num_ct1, float4Desc_x_ct1, float4Desc_y_ct1)));
    funcT(hipArrayCreate(&a42, &float4Desc));
    // CHECK: funcT(DPCT_CHECK_ERROR(delete a42));
    funcT(hipArrayDestroy(a42));
  }
}

void create_array_fail() {
  hipArray_t a;
  unsigned i;
  // CHECK: CUDA_ARRAY_DESCRIPTOR d[20], *p;
  HIP_ARRAY_DESCRIPTOR d[20], *p;
  p = &d[5];

  // CHECK: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'd' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, d);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'p' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, p);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter 'p + i' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, p + i);
  // CHECK-NEXT: /*
  // CHECK-NEXT:  DPCT1073:{{[0-9]+}}: The field values of parameter '&d[i]' could not be deduced, so the call was not migrated. You need to update this code manually.
  // CHECK-NEXT: */
  // CHECK-NEXT: cuArrayCreate(&a, &d[i]);
  hipArrayCreate(&a, d);
  hipArrayCreate(&a, p);
  hipArrayCreate(&a, p + i);
  hipArrayCreate(&a, &d[i]);
}

void test_texref() {
  // CHECK: sycl::addressing_mode addr_mode;
  // CHECK-NEXT: sycl::filtering_mode filter_mode;
  // CHECK-NEXT: sycl::image_channel_type format;
  // CHECK-NEXT: dpct::image_matrix_p arr;
  // CHECK-NEXT: dpct::image_wrapper_base_p tex;
  // CHECK-NEXT: int err_code;
  HIPaddress_mode addr_mode;
  HIPfilter_mode filter_mode;
  hipArray_Format format;
  hipArray_t arr;
  hipTexRef tex;
  hipError_t err_code;
  int flags, chn_num;

  // CHECK: tex->set_channel_type(format);
  // CHECK-NEXT: tex->set_channel_num(4);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR((tex->set_channel_type(sycl::image_channel_type::fp32), tex->set_channel_num(chn_num)));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  hipTexRefSetFormat(tex, format, 4);
  err_code = hipTexRefSetFormat(tex, HIP_AD_FORMAT_FLOAT, chn_num);
  cudaCheck(hipTexRefSetFormat(tex, format, 4));
  func(hipTexRefSetFormat(tex,format,4));
  funcT(hipTexRefSetFormat(tex,format,4));

  // CHECK: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: tex->set_coordinate_normalization_mode(flags & 0x02);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::unnormalized)));
  hipTexRefSetFlags(tex, flags);
  err_code = hipTexRefSetFlags(tex, HIP_TRSF_NORMALIZED_COORDINATES);
  cudaCheck(hipTexRefSetFlags(tex,  HIP_TRSF_NORMALIZED_COORDINATES | HIP_TRSF_READ_AS_INTEGER));
  func(hipTexRefSetFlags(tex,3));
  funcT(hipTexRefSetFlags(tex,1));
  unsigned int uflag;
  // CHECK: uflag = tex->is_coordinate_normalized() << 1;
  hipTexRefGetFlags(&uflag, tex);

  // CHECK: tex->set(addr_mode);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::addressing_mode::clamp_to_edge));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  hipTexRefSetAddressMode(tex, 0, addr_mode);
  err_code = hipTexRefSetAddressMode(tex, 1, HIP_TR_ADDRESS_MODE_CLAMP);
  cudaCheck(hipTexRefSetAddressMode(tex, 2, addr_mode));
  func(hipTexRefSetAddressMode(tex,0,addr_mode));
  funcT(hipTexRefSetAddressMode(tex,0,addr_mode));

  // CHECK: addr_mode = tex->get_addressing_mode();
  hipTexRefGetAddressMode(&addr_mode, tex, 0);

  // CHECK: tex->set(filter_mode);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::filtering_mode::linear));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  hipTexRefSetFilterMode(tex, filter_mode);
  err_code = hipTexRefSetFilterMode(tex, HIP_TR_FILTER_MODE_LINEAR);
  cudaCheck(hipTexRefSetFilterMode(tex, filter_mode));
  func(hipTexRefSetFilterMode(tex,filter_mode));
  funcT(hipTexRefSetFilterMode(tex,filter_mode));

  // CHECK: filter_mode = tex->get_filtering_mode();
  hipTexRefGetFilterMode(&filter_mode, tex);

  // CHECK: tex->attach(dpct::image_data(arr));
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr)));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT);
  err_code = hipTexRefSetArray(tex, arr, 0x01);
  cudaCheck(hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT));
  func(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));
  funcT(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));

  // CHECK: dpct::device_ptr dptr;
  // CHECK-Next: size_t s, b;
  // CHECK-Next: tex->attach(dptr, b);
  // CHECK-Next: size_t desc_x_ct1, desc_y_ct1;
  // CHECK-Next: unsigned desc_channel_num_ct1;
  // CHECK-Next: sycl::image_channel_type desc_channel_type_ct1;
  // CHECK-Next: tex->attach(dptr, desc_x_ct1, desc_y_ct1, b);
  hipDeviceptr_t dptr;
  size_t s, b;
  hipTexRefSetAddress(&s, tex, dptr, b);
  HIP_ARRAY_DESCRIPTOR desc;
  hipTexRefSetAddress2D(tex, &desc, dptr, b);
}
