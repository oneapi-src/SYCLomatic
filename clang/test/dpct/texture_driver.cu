// RUN: dpct --format-range=none --usm-level=none -out-root %T/texture_driver %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture_driver/texture_driver.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/texture_driver/texture_driver.dp.cpp -o %T//texture_driver/texture_driver.dp.o %}

#include "hip/hip_runtime.h"
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  int err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

void func(int i) {}

template <typename T>
void funcT(T t) {}

int main() {
  // CHECK: dpct::image_matrix_desc p3DDesc;
  // CHECK-NEXT: p3DDesc.width = 1;
  // CHECK-NEXT: p3DDesc.height = 2;
  // CHECK-NEXT: p3DDesc.depth = 3;
  // CHECK-NEXT: p3DDesc.channel_type = sycl::image_channel_type::signed_int8;
  // CHECK-NEXT: p3DDesc.num_channels = 4;
  HIP_ARRAY3D_DESCRIPTOR p3DDesc;
  p3DDesc.Width = 1;
  p3DDesc.Height = 2;
  p3DDesc.Depth = 3;
  p3DDesc.Format = HIP_AD_FORMAT_SIGNED_INT8;
  p3DDesc.Flags = 5;
  p3DDesc.NumChannels = 4;

  // CHECK: dpct::image_matrix_desc halfDesc;
  // CHECK-NEXT: halfDesc.height = 32;
  // CHECK-NEXT: halfDesc.width = 64;
  // CHECK-NEXT: halfDesc.channel_type = sycl::image_channel_type::fp16;
  // CHECK-NEXT: halfDesc.num_channels = 1;
  HIP_ARRAY_DESCRIPTOR halfDesc;
  halfDesc.Height = 32;
  halfDesc.Width = 64;
  halfDesc.Format = HIP_AD_FORMAT_HALF;
  halfDesc.NumChannels = 1;

  // CHECK: dpct::image_matrix_desc float4Desc;
  // CHECK-NEXT: float4Desc.width = 64;
  // CHECK-NEXT: float4Desc.channel_type = sycl::image_channel_type::fp32;
  // CHECK-NEXT: float4Desc.num_channels = 4;
  // CHECK-NEXT: float4Desc.height = 32;
  HIP_ARRAY_DESCRIPTOR float4Desc;
  float4Desc.Width = 64;
  float4Desc.Format = HIP_AD_FORMAT_FLOAT;
  float4Desc.NumChannels = 4;
  float4Desc.Height = 32;

  // CHECK: dpct::image_matrix **a3d_ptr = new dpct::image_matrix_p;
  // CHECK-NEXT: *a3d_ptr = new dpct::image_matrix(&p3DDesc);
  // CHECK-NEXT: delete (*a3d_ptr);
  // CHECK-NEXT: delete a3d_ptr;
  hipArray **a3d_ptr = new hipArray_t;
  hipArray3DCreate(a3d_ptr, &p3DDesc);
  hipArrayDestroy(*a3d_ptr);
  delete a3d_ptr;

  // CHECK: dpct::image_matrix **a_ptr = new dpct::image_matrix_p;
  // CHECK-NEXT: dpct::image_matrix_p a42;
  // CHECK-NEXT: *a_ptr = new dpct::image_matrix(&halfDesc);
  // CHECK-NEXT: a42 = new dpct::image_matrix(&float4Desc);
  // CHECK-NEXT: delete (*a_ptr);
  // CHECK-NEXT: delete a42;
  // CHECK-NEXT: delete a_ptr;
  hipArray **a_ptr = new hipArray_t;
  hipArray_t a42;
  hipArrayCreate(a_ptr, &halfDesc);
  hipArrayCreate(&a42, &float4Desc);
  hipArrayDestroy(*a_ptr);
  hipArrayDestroy(a42);
  delete a_ptr;

  // Test IsAssigned
  {
    int errorCode;

    // CHECK: errorCode = DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(&float4Desc));
    errorCode = hipArrayCreate(&a42, &float4Desc);
    // CHECK: errorCode = DPCT_CHECK_ERROR(delete a42);
    errorCode = hipArrayDestroy(a42);


    // CHECK: cudaCheck(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(&float4Desc)));
    cudaCheck(hipArrayCreate(&a42, &float4Desc));
    // CHECK: cudaCheck(DPCT_CHECK_ERROR(delete a42));
    cudaCheck(hipArrayDestroy(a42));


    // CHECK: func(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(&float4Desc)));
    func(hipArrayCreate(&a42, &float4Desc));
    // CHECK: func(DPCT_CHECK_ERROR(delete a42));
    func(hipArrayDestroy(a42));


    // CHECK: funcT(DPCT_CHECK_ERROR(a42 = new dpct::image_matrix(&float4Desc)));
    funcT(hipArrayCreate(&a42, &float4Desc));
    // CHECK: funcT(DPCT_CHECK_ERROR(delete a42));
    funcT(hipArrayDestroy(a42));
  }
}

void create_array_fail() {
  hipArray_t a;
  unsigned i;
  // CHECK: dpct::image_matrix_desc d[20], *p;
  HIP_ARRAY_DESCRIPTOR d[20], *p;
  p = &d[5];

  // CHECK: a = new dpct::image_matrix(d);
  // CHECK-NEXT: a = new dpct::image_matrix(p);
  // CHECK-NEXT: a = new dpct::image_matrix(p + i);
  // CHECK-NEXT: a = new dpct::image_matrix(&d[i]);
  hipArrayCreate(&a, d);
  hipArrayCreate(&a, p);
  hipArrayCreate(&a, p + i);
  hipArrayCreate(&a, &d[i]);
}

void test_texref() {
  // CHECK: sycl::addressing_mode addr_mode;
  // CHECK-NEXT: sycl::filtering_mode filter_mode;
  // CHECK-NEXT: sycl::image_channel_type format;
  // CHECK-NEXT: dpct::image_matrix_p arr;
  // CHECK-NEXT: dpct::image_wrapper_base_p tex;
  // CHECK-NEXT: int err_code;
  HIPaddress_mode addr_mode;
  HIPfilter_mode filter_mode;
  hipArray_Format format;
  hipArray_t arr;
  hipTexRef tex;
  hipError_t err_code;
  int flags, chn_num;

  // CHECK: tex->set_channel_type(format);
  // CHECK-NEXT: tex->set_channel_num(4);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR((tex->set_channel_type(sycl::image_channel_type::fp32), tex->set_channel_num(chn_num)));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR((tex->set_channel_type(format), tex->set_channel_num(4))));
  hipTexRefSetFormat(tex, format, 4);
  err_code = hipTexRefSetFormat(tex, HIP_AD_FORMAT_FLOAT, chn_num);
  cudaCheck(hipTexRefSetFormat(tex, format, 4));
  func(hipTexRefSetFormat(tex,format,4));
  funcT(hipTexRefSetFormat(tex,format,4));

  // CHECK: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: tex->set_coordinate_normalization_mode(flags & 0x02);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::normalized)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(sycl::coordinate_normalization_mode::unnormalized)));
  hipTexRefSetFlags(tex, flags);
  err_code = hipTexRefSetFlags(tex, HIP_TRSF_NORMALIZED_COORDINATES);
  cudaCheck(hipTexRefSetFlags(tex,  HIP_TRSF_NORMALIZED_COORDINATES | HIP_TRSF_READ_AS_INTEGER));
  func(hipTexRefSetFlags(tex,3));
  funcT(hipTexRefSetFlags(tex,1));
  unsigned int uflag;
  // CHECK: uflag = tex->is_coordinate_normalized() << 1;
  hipTexRefGetFlags(&uflag, tex);

  // CHECK: tex->set(addr_mode);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::addressing_mode::clamp_to_edge));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(addr_mode)));
  hipTexRefSetAddressMode(tex, 0, addr_mode);
  err_code = hipTexRefSetAddressMode(tex, 1, HIP_TR_ADDRESS_MODE_CLAMP);
  cudaCheck(hipTexRefSetAddressMode(tex, 2, addr_mode));
  func(hipTexRefSetAddressMode(tex,0,addr_mode));
  funcT(hipTexRefSetAddressMode(tex,0,addr_mode));

  // CHECK: addr_mode = tex->get_addressing_mode();
  hipTexRefGetAddressMode(&addr_mode, tex, 0);

  // CHECK: tex->set(filter_mode);
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->set(sycl::filtering_mode::linear));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->set(filter_mode)));
  hipTexRefSetFilterMode(tex, filter_mode);
  err_code = hipTexRefSetFilterMode(tex, HIP_TR_FILTER_MODE_LINEAR);
  cudaCheck(hipTexRefSetFilterMode(tex, filter_mode));
  func(hipTexRefSetFilterMode(tex,filter_mode));
  funcT(hipTexRefSetFilterMode(tex,filter_mode));

  // CHECK: filter_mode = tex->get_filtering_mode();
  hipTexRefGetFilterMode(&filter_mode, tex);

  // CHECK: tex->attach(dpct::image_data(arr));
  // CHECK-NEXT: err_code = DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr)));
  // CHECK-NEXT: cudaCheck(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  // CHECK-NEXT: func(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  // CHECK-NEXT: funcT(DPCT_CHECK_ERROR(tex->attach(dpct::image_data(arr))));
  hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT);
  err_code = hipTexRefSetArray(tex, arr, 0x01);
  cudaCheck(hipTexRefSetArray(tex, arr, HIP_TRSA_OVERRIDE_FORMAT));
  func(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));
  funcT(hipTexRefSetArray(tex,arr, HIP_TRSA_OVERRIDE_FORMAT));

  // CHECK: dpct::device_ptr dptr;
  // CHECK-Next: size_t s, b;
  // CHECK-Next: tex->attach(dptr, b);
  // CHECK-Next: size_t desc_x_ct1, desc_y_ct1;
  // CHECK-Next: unsigned desc_channel_num_ct1;
  // CHECK-Next: sycl::image_channel_type desc_channel_type_ct1;
  // CHECK-Next: tex->attach(dptr, desc_x_ct1, desc_y_ct1, b);
  hipDeviceptr_t dptr;
  size_t s, b;
  hipTexRefSetAddress(&s, tex, dptr, b);
  HIP_ARRAY_DESCRIPTOR desc;
  hipTexRefSetAddress2D(tex, &desc, dptr, b);
}

// CHECK: sycl::addressing_mode AddrMode[] =
// CHECK-NEXT: {
// CHECK-NEXT:   sycl::addressing_mode::repeat,
// CHECK-NEXT:   sycl::addressing_mode::clamp_to_edge,
// CHECK-NEXT:   sycl::addressing_mode::clamp
// CHECK-NEXT: };
HIPaddress_mode AddrMode[] =
{
  HIP_TR_ADDRESS_MODE_WRAP,
  HIP_TR_ADDRESS_MODE_CLAMP,
  HIP_TR_ADDRESS_MODE_BORDER
};

// CHECK: sycl::filtering_mode FltMode[] =
// CHECK-NEXT: {
// CHECK-NEXT:   sycl::filtering_mode::nearest,
// CHECK-NEXT:   sycl::filtering_mode::linear
// CHECK-NEXT: };
HIPfilter_mode FltMode[] =
{
  HIP_TR_FILTER_MODE_POINT,
  HIP_TR_FILTER_MODE_LINEAR
};

// CHECK: void TestAssignment(sycl::addressing_mode a) {
// CHECK-NEXT:   if (a == sycl::addressing_mode::repeat);
// CHECK-NEXT:   if (a == sycl::addressing_mode::clamp_to_edge);
// CHECK-NEXT: }
void TestAssignment(HIPaddress_mode a) {
  if (a == HIP_TR_ADDRESS_MODE_WRAP);
  if (a == HIP_TR_ADDRESS_MODE_CLAMP);
}
