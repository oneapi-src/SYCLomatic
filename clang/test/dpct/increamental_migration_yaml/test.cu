
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --out-root %T/increamental_migration_yaml/out %s --use-custom-helper=api --custom-helper-name=test --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -DAAA
// RUN: dpct --format-range=none --out-root %T/increamental_migration_yaml/out %s --use-custom-helper=api --custom-helper-name=test --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/increamental_migration_yaml/out/MainSourceFiles.yaml --match-full-lines %s
// RUN: rm -rf %T/increamental_migration_yaml/out

//     CHECK:FeatureMap:
//CHECK-NEXT:  test.hpp:
//CHECK-NEXT:    dpct_align_and_inline:
//CHECK-NEXT:      IsCalled:        true
//CHECK-NEXT:      CallerSrcFiles:
//CHECK-NEXT:        - '{{(.+)}}'
//CHECK-NEXT:      SubFeatureMap:   {}
//CHECK-NEXT:    dpct_compatibility_temp:
//CHECK-NEXT:      IsCalled:        true
//CHECK-NEXT:      CallerSrcFiles:
//CHECK-NEXT:        - '{{(.+)}}'
//CHECK-NEXT:      SubFeatureMap:   {}
//CHECK-NEXT:    non_local_include_dependency:
//CHECK-NEXT:      IsCalled:        true
//CHECK-NEXT:      CallerSrcFiles:
//CHECK-NEXT:        - ''
//CHECK-NEXT:      SubFeatureMap:   {}

#ifdef AAA
__forceinline__ void foo(){}
#else
#define BBB __CUDA_ARCH__
#endif

int main() {
  return 0;
}
