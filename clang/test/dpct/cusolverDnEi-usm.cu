// RUN: dpct --format-range=none -out-root %T/cusolverDnEi-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnEi-usm/cusolverDnEi-usm.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusolverDnEi-usm/cusolverDnEi-usm.dp.cpp -o %T/cusolverDnEi-usm/cusolverDnEi-usm.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipsolverEigMode_t jobz;

    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    signed char jobu;
    signed char jobvt;

    float S_f = 0;
    double S_d = 0.0;
    hipComplex S_c = make_hipComplex(1,0);
    hipDoubleComplex S_z = make_hipDoubleComplex(1,0);

    float U_f = 0;
    double U_d = 0.0;
    hipComplex U_c = make_hipComplex(1,0);
    hipDoubleComplex U_z = make_hipDoubleComplex(1,0);
    int ldu;

    float VT_f = 0;
    double VT_d = 0.0;
    hipComplex VT_c = make_hipComplex(1,0);
    hipDoubleComplex VT_z = make_hipDoubleComplex(1,0);
    int ldvt;

    float Rwork_f = 0;
    double Rwork_d = 0.0;
    hipComplex Rwork_c = make_hipComplex(1,0);
    hipDoubleComplex Rwork_z = make_hipDoubleComplex(1,0);

    float W_f = 0;
    double W_d = 0.0;
    hipComplex W_c = make_hipComplex(1,0);
    hipDoubleComplex W_z = make_hipDoubleComplex(1,0);


    //CHECK: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, (float*)&A_f, lda, (float*)&D_f, (float*)&E_f, (float*)&TAUQ_f, (float*)&TAUP_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, (double*)&A_d, lda, (double*)&D_d, (double*)&E_d, (double*)&TAUQ_d, (double*)&TAUP_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, (std::complex<float>*)&A_c, lda, (float*)&D_f, (float*)&E_f, (std::complex<float>*)&TAUQ_c, (std::complex<float>*)&TAUP_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, (std::complex<double>*)&A_z, lda, (double*)&D_d, (double*)&E_d, (std::complex<double>*)&TAUQ_z, (std::complex<double>*)&TAUP_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);

    //CHECK: oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, (float*)&A_f, lda, (float*)&TAU_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, (double*)&A_d, lda, (double*)&TAU_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, (float*)&A_f, lda, (float*)&D_f, (float*)&E_f, (float*)&TAU_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, (double*)&A_d, lda, (double*)&D_d, (double*)&E_d, (double*)&TAU_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, (float*)&D_f, (float*)&E_f, (std::complex<float>*)&TAU_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, (double*)&D_d, (double*)&E_d, (std::complex<double>*)&TAU_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, (float*)&A_f, lda, (float*)&TAU_f, (float*)&B_f, ldb, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, (double*)&A_d, lda, (double*)&TAU_d, (double*)&B_d, ldb, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&B_c, ldb, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&B_z, ldb, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    //CHECK: oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, (float*)&A_f, lda, (float*)&TAU_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, (double*)&A_d, lda, (double*)&TAU_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, (float*)&A_f, lda, (float*)&S_f, (float*)&U_f, ldu, (float*)&VT_f, ldvt, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, (double*)&A_d, lda, (double*)&S_d, (double*)&U_d, ldu, (double*)&VT_d, ldvt, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, (std::complex<float>*)&A_c, lda, (float*)&S_f, (std::complex<float>*)&U_c, ldu, (std::complex<float>*)&VT_c, ldvt, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, (std::complex<double>*)&A_z, lda, (double*)&S_d, (std::complex<double>*)&U_z, ldu, (std::complex<double>*)&VT_z, ldvt, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);
    hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);
    hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);
    hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);

}

