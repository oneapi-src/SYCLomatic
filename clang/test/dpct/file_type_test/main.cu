#include "hip/hip_runtime.h"
// UNSUPPORTED: system-windows
// RUN: echo "[" > %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/main.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/main.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/bar/util_bar.cc\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/bar/util_bar.cc\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/bar/util.gpu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/bar/util.gpu\"" >> %T/compile_commands.json
// RUN: echo "    }" >> %T/compile_commands.json
// RUN: echo "]" >> %T/compile_commands.json

// RUN: dpct --format-range=none --cuda-include-path="%cuda-path/include" -in-root=%S -out-root=%T -p=%T %s %S/bar/util.gpu %S/bar/util_bar.cc --sycl-named-lambda -extra-arg="-I%S/bar" -extra-arg="--std=c++11"
// RUN: FileCheck %s --match-full-lines --input-file %T/main.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/main.dp.cpp -o %T/main.dp.o %}
// RUN: FileCheck %S/bar/util.gpu --match-full-lines --input-file %T/bar/util.gpu.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/bar/util.gpu.dp.cpp -o %T/bar/util.gpu.dp.o %}
// RUN: FileCheck %S/bar/util.gpuhead --match-full-lines --input-file %T/bar/util.gpuhead
// RUN: FileCheck %S/bar/util_bar.hh --match-full-lines --input-file %T/bar/util_bar.hh
// RUN: FileCheck %S/bar/macro_def.hh --match-full-lines --input-file %T/bar/macro_def.hh
// RUN: FileCheck %S/bar/util_bar.cc --match-full-lines --input-file %T/bar/util_bar.cc.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/bar/util_bar.cc.dp.cpp -o %T/bar/util_bar.cc.dp.o %}

// RUN: dpct --format-range=none --cuda-include-path="%cuda-path/include" -in-root=%S -out-root=%T  -p=%T %S/main.gpu   --sycl-named-lambda
// RUN: FileCheck %S/main.gpu --match-full-lines --input-file %T/main.gpu.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/main.gpu.dp.cpp -o %T/main.gpu.dp.o %}

#ifndef  BUILD_TEST
#include <stdio.h>
#include <hip/hip_runtime.h>

// CHECK:#include "util.gpuhead"
#include "util.gpuhead"

// CHECK:#include "util_bar.hh"
#include "util_bar.hh"

// CHECK:void FooKernel() {
__global__ void FooKernel() {
   foo_util();
   util_bar();
}

int main(){
 int *a, *b;
 hipMalloc((void **)&a, 4);
 hipMalloc((void **)&b, 4);
 kernel_util<<<1,1>>>(a,b);
 FooKernel<<<1,1>>>();
}
#endif
