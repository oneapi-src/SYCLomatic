// RUN: dpct --format-range=none --report-type=all -out-root %T/cusolverHelper %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverHelper/cusolverHelper.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusolverHelper/cusolverHelper.dp.cpp -o %T/cusolverHelper/cusolverHelper.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>

#include <hip/hip_runtime.h>
#include <hipsolver.h>

// CHECK: #define MACRO_A cusolverDnCreate
#define MACRO_A hipsolverDnCreate

// CHECK: void foo(int, int, int, int, int, int, int, int) {}
void foo(hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t, hipsolverStatus_t) {}

// CHECK: void foo2(int){}
void foo2(hipsolverStatus_t){}

// CHECK: int foo3(int m, int n)
hipsolverStatus_t foo3(int m, int n)
{
    // CHECK: return 0;
    return HIPSOLVER_STATUS_SUCCESS;
}

// CHECK: extern sycl::queue* cusolverH2 = NULL;
extern hipsolverHandle_t cusolverH2 = NULL;

int main(int argc, char *argv[])
{
    // CHECK: sycl::queue* cusolverH = NULL;
    // CHECK-NEXT: int status = 0;
    // CHECK-NEXT: status = 1;
    hipsolverHandle_t cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;

    // CHECK: foo(0, 1, 2, 3, 4, 6, 7, 8);
    foo(HIPSOLVER_STATUS_SUCCESS, HIPSOLVER_STATUS_NOT_INITIALIZED, HIPSOLVER_STATUS_ALLOC_FAILED, HIPSOLVER_STATUS_INVALID_VALUE, HIPSOLVER_STATUS_ARCH_MISMATCH, HIPSOLVER_STATUS_EXECUTION_FAILED, HIPSOLVER_STATUS_INTERNAL_ERROR, HIPSOLVER_STATUS_MATRIX_TYPE_NOT_SUPPORTED);

    // CHECK: cusolverH = &q_ct1;
    hipsolverDnCreate(&cusolverH);

    // CHECK: status = DPCT_CHECK_ERROR(cusolverH = &q_ct1);
    status = hipsolverDnCreate(&cusolverH);

    // CHECK: status = DPCT_CHECK_ERROR(cusolverH = &q_ct1);
    status = MACRO_A(&cusolverH);

    // CHECK: status = DPCT_CHECK_ERROR(cusolverH = nullptr);
    status = hipsolverDnDestroy(cusolverH);

    // CHECK: int a = sizeof(int);
    // CHECK-NEXT: int b = sizeof(sycl::queue*);
    int a = sizeof(hipblasStatus_t);
    int b = sizeof(hipsolverHandle_t);

    hipStream_t stream;
    // CHECK: status = DPCT_CHECK_ERROR(stream = cusolverH);
    // CHECK: status = DPCT_CHECK_ERROR(cusolverH = stream);
    status = hipsolverGetStream(cusolverH, &stream);
    status = hipsolverSetStream(cusolverH, stream);
}


