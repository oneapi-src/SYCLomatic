
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/sizeof_int2_insert_namespace %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/sizeof_int2_insert_namespace/sizeof_int2_insert_namespace.dp.cpp --match-full-lines %s

void fun() {
  // CHECK:  sycl::mint2 a, b, c, d[2], *e[2];
  int2 a, b, c, d[2], *e[2];
  // CHECK:  int i = sizeof(sycl::mint2);
  int i = sizeof(int2);
  // CHECK:  int j = sizeof(int);
  int j = sizeof(int);
  // CHECK:  sycl::mint2 k;
  int2 k;
  // CHECK:  int kk = sizeof(k);
  int kk = sizeof(k);
}

