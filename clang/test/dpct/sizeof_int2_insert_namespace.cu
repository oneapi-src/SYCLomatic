
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/sizeof_int2_insert_namespace %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/sizeof_int2_insert_namespace/sizeof_int2_insert_namespace.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/sizeof_int2_insert_namespace/sizeof_int2_insert_namespace.dp.cpp -o %T/sizeof_int2_insert_namespace/sizeof_int2_insert_namespace.dp.o %}

void fun() {
  // CHECK:  sycl::int2 a, b, c, d[2], *e[2];
  int2 a, b, c, d[2], *e[2];
  // CHECK:  int i = sizeof(sycl::int2);
  int i = sizeof(int2);
  // CHECK:  int j = sizeof(int);
  int j = sizeof(int);
  // CHECK:  sycl::int2 k;
  int2 k;
  // CHECK:  int kk = sizeof(k);
  int kk = sizeof(k);
}

