#include "hip/hip_runtime.h"
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none -out-root %T/compat_nvcc %s --cuda-include-path="%cuda-path/include" --stop-on-parse-err
// RUN: FileCheck %s --match-full-lines --input-file %T/compat_nvcc/compat_nvcc.dp.cpp

#include "hip/hip_fp16.h"

// CHECK: inline void ff(sycl::half2 *array, sycl::half a) {
// CHECK-NEXT:   array[10] = {a, a};
// CHECK-NEXT: }
__device__ inline void ff(__half2 *array, __half a) {
  array[10] = {a, a};
}
