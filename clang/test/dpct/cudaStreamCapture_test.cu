// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --use-experimental-features=graph --format-range=none -out-root %T/cudaStreamCapture_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cudaStreamCapture_test/cudaStreamCapture_test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cudaStreamCapture_test/cudaStreamCapture_test.dp.cpp -o %T/cudaStreamCapture_test/cudaStreamCapture_test.dp.o %}

#include <hip/hip_runtime.h>
#define CUDA_CHECK_THROW(x)  \
  do {                       \
    hipError_t _result = x; \
  } while (0)

int main() {
  hipGraph_t graph;
  hipGraph_t *graph2;
  hipGraph_t **graph3;

  hipStream_t stream;
  hipStreamCreate(&stream);

  // CHECK: dpct::experimental::begin_recording(stream);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(dpct::experimental::begin_recording(stream)));
  // CHECK-NEXT: dpct::experimental::end_recording(stream, &graph);
  // CHECK-NEXT: dpct::experimental::end_recording(stream, graph2);
  // CHECK-NEXT: dpct::experimental::end_recording(stream, *graph3);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(dpct::experimental::end_recording(stream, *graph3)));
  hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal);
  CUDA_CHECK_THROW(hipStreamBeginCapture(stream, hipStreamCaptureModeGlobal));
  hipStreamEndCapture(stream, &graph);
  hipStreamEndCapture(stream, graph2);
  hipStreamEndCapture(stream, *graph3);
  CUDA_CHECK_THROW(hipStreamEndCapture(stream, *graph3));

  hipStreamCaptureStatus captureStatus;
  hipStreamCaptureStatus *captureStatus2;


  // CHECK: captureStatus = stream->ext_oneapi_get_state();
  // CHECK: *captureStatus2 = q_ct1.ext_oneapi_get_state();
  // CHECK: *captureStatus2 = q_ct1.ext_oneapi_get_state();
  hipStreamIsCapturing(stream, &captureStatus);
  hipStreamIsCapturing(hipStreamLegacy, captureStatus2);
  hipStreamIsCapturing(hipStreamDefault, captureStatus2);

  return 0;
}
