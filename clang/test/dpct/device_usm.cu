// RUN: dpct --format-range=none -out-root %T/device_usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/device_usm/device_usm.dp.cpp
#include <hip/hip_runtime.h>

int main() {
int concurrentManagedAccess = 0;
int p_gpuDevice = 0;
// CHECK: int error = (concurrentManagedAccess = dpct::dev_mgr::instance().get_device(p_gpuDevice).get_info<sycl::info::device::usm_shared_allocations>(), 0);
int error = hipDeviceGetAttribute(&concurrentManagedAccess,  hipDeviceAttributeConcurrentManagedAccess,  p_gpuDevice);
}