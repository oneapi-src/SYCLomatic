// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0 
// RUN: dpct --format-range=none -out-root %T/cooperative_groups_reduce %s --cuda-include-path="%cuda-path/include" --extra-arg="-std=c++14"
// RUN: FileCheck %s --match-full-lines --input-file %T/cooperative_groups_reduce/cooperative_groups_reduce.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/cooperative_groups_reduce/cooperative_groups_reduce.dp.cpp -o %T/cooperative_groups_reduce/cooperative_groups_reduce.dp.o %}


#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>

namespace cg = cooperative_groups;

__device__ void testReduce(double *sdata, const cg::thread_block &cta) {
  const unsigned int tid = cta.thread_rank();
  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);
  int *idata;
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), sdata[tid], sycl::plus<double>());
  cg::reduce(tile32, sdata[tid], cg::plus<double>());  
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), sdata[tid], sycl::minimum<double>());
  cg::reduce(tile32, sdata[tid], cg::less<double>());  
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), sdata[tid], sycl::maximum<double>());
  cg::reduce(tile32, sdata[tid], cg::greater<double>());  
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), idata[tid], sycl::bit_and<int>());
  cg::reduce(tile32, idata[tid], cg::bit_and<int>());  
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), idata[tid], sycl::bit_xor<int>());
  cg::reduce(tile32, idata[tid], cg::bit_xor<int>());
  // CHECK: sycl::reduce_over_group(item_ct1.get_sub_group(), idata[tid], sycl::bit_or<int>());
  cg::reduce(tile32, idata[tid], cg::bit_or<int>());
  cg::sync(cta);

}
