// UNSUPPORTED: -linux-
// RUN: mkdir %T/test_helpapi_stats_with_replacetext_windows
// RUN: cd %T/test_helpapi_stats_with_replacetext_windows
// RUN: dpct --format-range=none -out-root %T/test_helpapi_stats_with_replacetext_windows %s --cuda-include-path="%cuda-path/include"  --report-type=stats -- -x cuda --cuda-host-only > stats.txt
// RUN: echo "// CHECK: File name, LOC migrated to SYCL, LOC migrated to helper functions, LOC not needed to migrate, LOC not able to migrate" > %T/test_helpapi_stats_with_replacetext_windows/test_helpapi_stats_with_replacetext_ref.txt
// RUN: echo "// CHECK-NEXT: {{(.+)}}\test_helpapi_stats_with_replacetext_windows.cu, 14, 16, 47, 0" >> %T/test_helpapi_stats_with_replacetext_windows/test_helpapi_stats_with_replacetext_ref.txt
// RUN: FileCheck --match-full-lines --input-file %T/test_helpapi_stats_with_replacetext_windows/stats.txt %T/test_helpapi_stats_with_replacetext_windows/test_helpapi_stats_with_replacetext_ref.txt
// RUN: cd ..
// RUN: rm -rf ./test_helpapi_stats_with_replacetext_windows

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int nrhs = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;

  float **Aarray_S = 0;
  double **Aarray_D = 0;
  hipComplex **Aarray_C = 0;
  hipDoubleComplex **Aarray_Z = 0;

  float **Barray_S = 0;
  double **Barray_D = 0;
  hipComplex **Barray_C = 0;
  hipDoubleComplex **Barray_Z = 0;

  float **Carray_S = 0;
  double **Carray_D = 0;
  hipComplex **Carray_C = 0;
  hipDoubleComplex **Carray_Z = 0;

  float **TauArray_S = 0;
  double **TauArray_D = 0;
  hipComplex **TauArray_C = 0;
  hipDoubleComplex **TauArray_Z = 0;

  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;

  const float **Aarray_Sc = 0;
  const double **Aarray_Dc = 0;
  const hipComplex **Aarray_Cc = 0;
  const hipDoubleComplex **Aarray_Zc = 0;

  //The following 16 APIs are migrated to helper functions. The number "16" is this test focuses on.
  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);
  hipblasDgetrfBatched(handle, n, Aarray_D, lda, PivotArray, infoArray, batchSize);
  hipblasCgetrfBatched(handle, n, Aarray_C, lda, PivotArray, infoArray, batchSize);
  hipblasZgetrfBatched(handle, n, Aarray_Z, lda, PivotArray, infoArray, batchSize);
  hipblasSgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);
  hipblasDgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);
  hipblasCgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);
  hipblasZgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);
  hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);
  hipblasDgetriBatched(handle, n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);
  hipblasCgetriBatched(handle, n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);
  hipblasZgetriBatched(handle, n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);
  hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  hipblasDgeqrfBatched(handle, m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);
  hipblasCgeqrfBatched(handle, m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);
  hipblasZgeqrfBatched(handle, m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);

  return 0;
}
