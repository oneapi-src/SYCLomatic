// RUN: dpct --format-range=none -out-root %T/cufft-refine-setstream %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-refine-setstream/cufft-refine-setstream.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void foo1() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}


#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err = stmt;                                               \
  if (err != HIPFFT_SUCCESS) {                                           \
  }                                                                     \
} while(0)
void foo2() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck



#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line )
{
    if (err != HIPFFT_SUCCESS)
    {
      int a = err;
    }
}

void foo3() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:HANDLE_CUFFT_ERROR((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  HANDLE_CUFFT_ERROR(hipfftSetStream(plan, s));
  HANDLE_CUFFT_ERROR(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef HANDLE_CUFFT_ERROR


void foo4() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int err = (plan->set_queue(s), 0);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:err = (plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftResult err = hipfftSetStream(plan, s);
  err = hipfftExecR2C(plan, (float*)iodata, iodata);
}


static inline void CUFFT_CHECK(hipfftResult error)
{
  if (error != HIPFFT_SUCCESS) {
  }
}

void foo5() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:CUFFT_CHECK((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:CUFFT_CHECK((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CUFFT_CHECK(hipfftSetStream(plan, s));
  CUFFT_CHECK(hipfftExecR2C(plan, (float*)iodata, iodata));
}


#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}

void foo6() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:MY_ERROR_CHECKER((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:MY_ERROR_CHECKER((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  MY_ERROR_CHECKER(hipfftSetStream(plan, s));
  MY_ERROR_CHECKER(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef MY_ERROR_CHECKER


#define CHECK_CUFFT(call)                                                      \
{                                                                              \
  hipfftResult err;                                                           \
  if ( (err = (call)) != HIPFFT_SUCCESS)                                      \
  {                                                                          \
  }                                                                          \
}
void foo7() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:CHECK_CUFFT((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:CHECK_CUFFT((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CHECK_CUFFT(hipfftSetStream(plan, s));
  CHECK_CUFFT(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef CHECK_CUFFT

#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err;                                                      \
  if ( (err = (stmt)) != HIPFFT_SUCCESS) {                               \
  }                                                                     \
} while(0)
void foo8() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->set_queue(s), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}


void foo9() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s1, s2;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->set_queue(s1), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->set_queue(s2), 0));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  cufftCheck(hipfftSetStream(plan, s1));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  cufftCheck(hipfftSetStream(plan, s2));
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}

// This case needs manual fix
void foo10(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  //CHECK-NEXT:if (flag) {
  //CHECK-NEXT:  /*
  //CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:  */
  //CHECK-NEXT:  cufftCheck((plan->set_queue(s), 0));
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cufftCheck((plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward), 0));
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  if (flag) {
    cufftCheck(hipfftSetStream(plan, s));
  }
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck

void foo11(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:if (flag)
  //CHECK-NEXT:  plan->set_queue(s);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  if (flag)
    hipfftSetStream(plan, s);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo12(hipfftHandle plan2) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan = plan2;
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  plan = plan2;
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void changeHandle(hipfftHandle &p);

void foo13() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:changeHandle(plan);
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  changeHandle(plan);
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo14() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_default_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(s);
  //CHECK-NEXT:plan = dpct::fft::fft_engine::create();
  //CHECK-NEXT:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  //CHECK-NEXT:dpct::fft::fft_engine::destroy(plan);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);
  hipfftCreate(&plan);
  hipfftExecR2C(plan, (float*)iodata, iodata);
  hipfftDestroy(plan);
}
