// RUN: dpct --format-range=none -out-root %T/cufft-refine-setstream %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-refine-setstream/cufft-refine-setstream.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void foo1() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, s);

  //CHECK:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);
}


#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err = stmt;                                               \
  if (err != HIPFFT_SUCCESS) {                                           \
  }                                                                     \
} while(0)
void foo2() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck



#define HANDLE_CUFFT_ERROR( err ) (CufftHandleError( err, __FILE__, __LINE__ ))
static void CufftHandleError( hipfftResult err, const char *file, int line )
{
    if (err != HIPFFT_SUCCESS)
    {
      int a = err;
    }
}

void foo3() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  HANDLE_CUFFT_ERROR(hipfftSetStream(plan, s));

  //CHECK:HANDLE_CUFFT_ERROR([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  HANDLE_CUFFT_ERROR(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef HANDLE_CUFFT_ERROR


void foo4() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftResult err = hipfftSetStream(plan, s);

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:err = (oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata), 0);
  //CHECK-NEXT:}
  err = hipfftExecR2C(plan, (float*)iodata, iodata);
}


static inline void CUFFT_CHECK(hipfftResult error)
{
  if (error != HIPFFT_SUCCESS) {
  }
}

void foo5() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CUFFT_CHECK(hipfftSetStream(plan, s));

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:CUFFT_CHECK((oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata), 0));
  CUFFT_CHECK(hipfftExecR2C(plan, (float*)iodata, iodata));
}


#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}

void foo6() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  MY_ERROR_CHECKER(hipfftSetStream(plan, s));

  //CHECK:MY_ERROR_CHECKER([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  MY_ERROR_CHECKER(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef MY_ERROR_CHECKER


#define CHECK_CUFFT(call)                                                      \
{                                                                              \
  hipfftResult err;                                                           \
  if ( (err = (call)) != HIPFFT_SUCCESS)                                      \
  {                                                                          \
  }                                                                          \
}
void foo7() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  CHECK_CUFFT(hipfftSetStream(plan, s));

  //CHECK:CHECK_CUFFT([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  CHECK_CUFFT(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef CHECK_CUFFT

#define cufftCheck(stmt) \
do {                                           \
  hipfftResult err;                                                      \
  if ( (err = (stmt)) != HIPFFT_SUCCESS) {                               \
  }                                                                     \
} while(0)
void foo8() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  cufftCheck(hipfftSetStream(plan, s));

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}


void foo9() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s1, s2;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(q_ct1);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
  
  cufftCheck(hipfftSetStream(plan, s1));

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s1);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));

  cufftCheck(hipfftSetStream(plan, s2));

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s2);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}

// This case needs manual fix
void foo10(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(q_ct1);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));

  if (flag) {
    cufftCheck(hipfftSetStream(plan, s));
  }

  //CHECK:cufftCheck([&](){
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(q_ct1);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}());
  cufftCheck(hipfftExecR2C(plan, (float*)iodata, iodata));
}
#undef cufftCheck

void foo11(bool flag) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  if (flag)
    hipfftSetStream(plan, s);

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(dpct::get_default_queue());
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo12(hipfftHandle plan2) {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  hipfftSetStream(plan, s);
  plan = plan2;

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void changeHandle(hipfftHandle &p);

void foo13() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  hipfftSetStream(plan, s);
  changeHandle(plan);

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);
}

void foo14() {
  hipfftHandle plan;
  float2* iodata;
  hipStream_t s;

  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  hipfftSetStream(plan, s);
  //CHECK: DPCT1026:{{[0-9]+}}: The call to cufftCreate was removed because the function call is redundant in SYCL.
  hipfftCreate(&plan);

  //CHECK:/*
  //CHECK-NEXT:DPCT1075:{{[0-9]+}}: Migration of cuFFT calls may be incorrect and require review.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan->commit(*s);
  //CHECK-NEXT:if ((void *)(float*)iodata == (void *)iodata) {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata);
  //CHECK-NEXT:} else {
  //CHECK-NEXT:oneapi::mkl::dft::compute_forward(*plan, (float*)iodata, (float*)iodata);
  //CHECK-NEXT:}
  hipfftExecR2C(plan, (float*)iodata, iodata);
  //CHECK: DPCT1026:{{[0-9]+}}: The call to cufftDestroy was removed because the function call is redundant in SYCL.
  hipfftDestroy(plan);
}
