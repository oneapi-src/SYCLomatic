// RUN: dpct --format-range=none -out-root %T/cublasReturnType %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasReturnType/cublasReturnType.dp.cpp --match-full-lines %s
#include <hipblas.h>
#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  // CHECK: return 0;
  return HIPBLAS_STATUS_SUCCESS;
}

// CHECK: dpct::blas::descriptor_ptr foo1(int m) {
hipblasHandle_t foo1(int m) {
  return 0;
}

// CHECK: sycl::float2 foo2(sycl::float2 m) {
hipComplex foo2(hipComplex m) {
  // CHECK: return sycl::float2(1, 0);
  return make_hipComplex(1, 0);
}

// CHECK: sycl::double2 foo3(sycl::double2 m) {
hipDoubleComplex foo3(hipDoubleComplex m) {
  // CHECK: return sycl::double2(1, 0);
  return make_hipDoubleComplex(1, 0);
}

// CHECK: oneapi::mkl::transpose foo4(oneapi::mkl::transpose m) {
hipblasOperation_t foo4(hipblasOperation_t m) {
  // CHECK: return oneapi::mkl::transpose::conjtrans;
  return HIPBLAS_OP_C;
}

// CHECK: oneapi::mkl::uplo foo5(oneapi::mkl::uplo m) {
hipblasFillMode_t foo5(hipblasFillMode_t m) {
  // CHECK: return oneapi::mkl::uplo::lower;
  return HIPBLAS_FILL_MODE_LOWER;
}

// CHECK: oneapi::mkl::side foo6(oneapi::mkl::side m) {
hipblasSideMode_t foo6(hipblasSideMode_t m) {
  // CHECK: return oneapi::mkl::side::right;
  return HIPBLAS_SIDE_RIGHT;
}

// CHECK: oneapi::mkl::diag foo7(oneapi::mkl::diag m) {
hipblasDiagType_t foo7(hipblasDiagType_t m) {
  // CHECK: return oneapi::mkl::diag::nonunit;
  return HIPBLAS_DIAG_NON_UNIT;
}

