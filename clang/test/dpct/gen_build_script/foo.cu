
#include <hip/hip_runtime.h>
// UNSUPPORTED: -windows-
// RUN: rm -rf %T/build %T/source && mkdir -p %T/build/objs
// RUN: mkdir -p %T/build/objs
// RUN: mkdir %T/source
// RUN: cat %s > %T/source/foo.cu
// RUN: cat %s > %T/source/bar.cu
// RUN: cd %T/build
// RUN: echo "[" > compile_commands.json
// RUN: echo "    {" >> compile_commands.json
// RUN: echo "        \"command\": \"nvcc -c -std=c++20 -o /%T/build/objs/foo.cu.o /%T/source/foo.cu\"," >> compile_commands.json
// RUN: echo "        \"directory\": \"/%T\"," >> compile_commands.json
// RUN: echo "        \"file\": \"/%T/source/foo.cu\"" >> compile_commands.json
// RUN: echo "    }," >> compile_commands.json
// RUN: echo "    {" >> compile_commands.json
// RUN: echo "        \"command\": \"g++ -c -std=c++11 -o /%T/build/objs/bar.cu.o /%T/source/bar.cu\"," >> compile_commands.json
// RUN: echo "        \"directory\": \"/%T\"," >> compile_commands.json
// RUN: echo "        \"file\": \"/%T/source/bar.cu\"" >> compile_commands.json
// RUN: echo "    }," >> compile_commands.json
// RUN: echo "    {" >> compile_commands.json
// RUN: echo "        \"command\": \"ld objs/foo.cu.o objs/bar.cu.o -o app\"," >> compile_commands.json
// RUN: echo "        \"directory\": \"/%T/build\"" >> compile_commands.json
// RUN: echo "    }" >> compile_commands.json
// RUN: echo "]" >> compile_commands.json
// RUN: cd %T
// RUN: dpct -in-root ./ -out-root out -p build/ -gen-build-script --cuda-include-path="%cuda-path/include"
// RUN: cat %S/Makefile.dpct.ref  >%T/Makefile.dpct.check
// RUN: cat %T/out/Makefile.dpct >> %T/Makefile.dpct.check
// RUN: FileCheck --match-full-lines --input-file %T/Makefile.dpct.check %T/Makefile.dpct.check

__global__ void foo() {
}

int main() {
	return 0;
}
