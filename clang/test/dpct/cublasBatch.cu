// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasBatch %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasBatch/cublasBatch.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublasBatch/cublasBatch.dp.cpp -o %T/cublasBatch/cublasBatch.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>
int main() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int nrhs = 275;
  int lda = 275;
  int ldb = 275;
  int ldc = 275;

  float **Aarray_S = 0;
  double **Aarray_D = 0;
  hipComplex **Aarray_C = 0;
  hipDoubleComplex **Aarray_Z = 0;

  float **Barray_S = 0;
  double **Barray_D = 0;
  hipComplex **Barray_C = 0;
  hipDoubleComplex **Barray_Z = 0;

  float **Carray_S = 0;
  double **Carray_D = 0;
  hipComplex **Carray_C = 0;
  hipDoubleComplex **Carray_Z = 0;

  float **TauArray_S = 0;
  double **TauArray_D = 0;
  hipComplex **TauArray_C = 0;
  hipDoubleComplex **TauArray_Z = 0;

  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;

  const float **Aarray_Sc = 0;
  const double **Aarray_Dc = 0;
  const hipComplex **Aarray_Cc = 0;
  const hipDoubleComplex **Aarray_Zc = 0;

  // getrf_batch
  // CHECK: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasSgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_S, lda, PivotArray, infoArray, batchSize));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasSgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_S, lda, PivotArray, infoArray, batchSize);
  status = hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);
  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);

  // CHECK: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasDgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_D, lda, PivotArray, infoArray, batchSize));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasDgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_D, lda, PivotArray, infoArray, batchSize);
  status = hipblasDgetrfBatched(handle, n, Aarray_D, lda, PivotArray, infoArray, batchSize);
  hipblasDgetrfBatched(handle, n, Aarray_D, lda, PivotArray, infoArray, batchSize);

  // CHECK: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasCgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_C, lda, PivotArray, infoArray, batchSize));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasCgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_C, lda, PivotArray, infoArray, batchSize);
  status = hipblasCgetrfBatched(handle, n, Aarray_C, lda, PivotArray, infoArray, batchSize);
  hipblasCgetrfBatched(handle, n, Aarray_C, lda, PivotArray, infoArray, batchSize);

  // CHECK: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasZgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_Z, lda, PivotArray, infoArray, batchSize));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of PivotArray in the dpct::getrf_batch_wrapper is different from the cublasZgetrfBatched. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::getrf_batch_wrapper(handle->get_queue(), n, Aarray_Z, lda, PivotArray, infoArray, batchSize);
  status = hipblasZgetrfBatched(handle, n, Aarray_Z, lda, PivotArray, infoArray, batchSize);
  hipblasZgetrfBatched(handle, n, Aarray_Z, lda, PivotArray, infoArray, batchSize);

  // getrs_batch
  // CHECK: status = DPCT_CHECK_ERROR(dpct::getrs_batch_wrapper(handle->get_queue(), dpct::get_transpose(trans0), n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize));
  // CHECK-NEXT: dpct::getrs_batch_wrapper(handle->get_queue(), oneapi::mkl::transpose::nontrans, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);
  status = hipblasSgetrsBatched(handle, (hipblasOperation_t)trans0, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);
  hipblasSgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getrs_batch_wrapper(handle->get_queue(), dpct::get_transpose(trans1), n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize));
  // CHECK-NEXT: dpct::getrs_batch_wrapper(handle->get_queue(), oneapi::mkl::transpose::nontrans, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);
  status = hipblasDgetrsBatched(handle, (hipblasOperation_t)trans1, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);
  hipblasDgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Dc, lda, PivotArray, Barray_D, ldb, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getrs_batch_wrapper(handle->get_queue(), dpct::get_transpose(trans2), n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize));
  // CHECK-NEXT: dpct::getrs_batch_wrapper(handle->get_queue(), oneapi::mkl::transpose::nontrans, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);
  status = hipblasCgetrsBatched(handle, (hipblasOperation_t)trans2, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);
  hipblasCgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Cc, lda, PivotArray, Barray_C, ldb, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getrs_batch_wrapper(handle->get_queue(), oneapi::mkl::transpose::conjtrans, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize));
  // CHECK-NEXT: dpct::getrs_batch_wrapper(handle->get_queue(), oneapi::mkl::transpose::nontrans, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);
  status = hipblasZgetrsBatched(handle, (hipblasOperation_t)2, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);
  hipblasZgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Zc, lda, PivotArray, Barray_Z, ldb, infoArray, batchSize);

  // getri_batch
  // CHECK: status = DPCT_CHECK_ERROR(dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize));
  // CHECK-NEXT: dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);
  status = hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);
  hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize));
  // CHECK-NEXT: dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);
  status = hipblasDgetriBatched(handle, n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);
  hipblasDgetriBatched(handle, n, Aarray_Dc, lda, PivotArray, Carray_D, ldc, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize));
  // CHECK-NEXT: dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);
  status = hipblasCgetriBatched(handle, n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);
  hipblasCgetriBatched(handle, n, Aarray_Cc, lda, PivotArray, Carray_C, ldc, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize));
  // CHECK-NEXT: dpct::getri_batch_wrapper(handle->get_queue(), n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);
  status = hipblasZgetriBatched(handle, n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);
  hipblasZgetriBatched(handle, n, Aarray_Zc, lda, PivotArray, Carray_Z, ldc, infoArray, batchSize);

  // geqrf_batch
  // CHECK: status = DPCT_CHECK_ERROR(dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize));
  // CHECK-NEXT: dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  status = hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize));
  // CHECK-NEXT: dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);
  status = hipblasDgeqrfBatched(handle, m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);
  hipblasDgeqrfBatched(handle, m, n, Aarray_D, lda, TauArray_D, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize));
  // CHECK-NEXT: dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);
  status = hipblasCgeqrfBatched(handle, m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);
  hipblasCgeqrfBatched(handle, m, n, Aarray_C, lda, TauArray_C, infoArray, batchSize);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize));
  // CHECK-NEXT: dpct::geqrf_batch_wrapper(handle->get_queue(), m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);
  status = hipblasZgeqrfBatched(handle, m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);
  hipblasZgeqrfBatched(handle, m, n, Aarray_Z, lda, TauArray_Z, infoArray, batchSize);

  return 0;
}
