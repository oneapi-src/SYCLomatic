#include "hip/hip_runtime.h"
// FIXME
// UNSUPPORTED: system-windows
// RUN: dpct --format-range=none --usm-level=restricted -out-root %T/USM-restricted %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-restricted/USM-restricted.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/USM-restricted/USM-restricted.dp.cpp -o %T/USM-restricted/USM-restricted.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <memory>
#include <vector>

#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)

__constant__ float constData[123 * 4];

int foo_b(int a){
  return 0;
}

void foo() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  hipStream_t stream;

  /// malloc
  // CHECK: d_A = (float *)sycl::malloc_device(size, q_ct1);
  hipMalloc((void **)&d_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(d_A = (float *)sycl::malloc_device(size, q_ct1));
  errorCode = hipMalloc((void **)&d_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(d_A = (float *)sycl::malloc_device(size, q_ct1)));
  MY_SAFE_CALL(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_device(sizeof(sycl::double2) + size, q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(sycl::uchar4) + size, q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_device(sizeof(d_A[0]), q_ct1);
  hipMalloc((void **)&d_A, sizeof(double2) + size);
  hipMalloc((void **)&d_A, sizeof(uchar4) + size);
  hipMalloc((void **)&d_A, sizeof(d_A[0]));

  // CHECK: d_A = (float *)dpct::dpct_malloc(size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK: p_A = dpct::dpct_malloc(e);
  hipMalloc3D(&p_A, e);

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: errorCode = DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1));
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1)));
  MY_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocDefault));

  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocDefault);
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocDefault);

  void *h_B = h_A;
  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostMalloc((void **)&h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1));
  errorCode = hipHostMalloc((void **)&h_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1)));
  MY_SAFE_CALL(hipHostMalloc((void **)&h_A, size));

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipMemAllocHost((void **)&h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1));
  errorCode = hipMemAllocHost((void **)&h_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1)));
  MY_SAFE_CALL(hipMemAllocHost((void **)&h_A, size));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) * size, q_ct1);
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) * size, q_ct1);
  hipHostMalloc((void **)&h_A, sizeof(double2) * size);
  hipHostMalloc((void **)&h_A, sizeof(uchar4) * size);

  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostMalloc(&h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1));
  errorCode = hipHostMalloc(&h_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1)));
  MY_SAFE_CALL(hipHostMalloc(&h_A, size));

  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) / size, q_ct1);
  // CHECK-NEXT: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) / size, q_ct1);
  hipHostMalloc(&h_A, sizeof(double2) / size);
  hipHostMalloc(&h_A, sizeof(uchar4) / size);

  float* buffer[2];
#define SIZE_1 (128 * 1024 * 1024)
  // CHECK: *buffer = sycl::malloc_host<float>(SIZE_1, q_ct1);
  // CHECK-NEXT: *(buffer + 1) = sycl::malloc_host<float>(SIZE_1, q_ct1);
  hipHostMalloc((void**)buffer, SIZE_1 * sizeof(float));
  hipHostMalloc((void**)(buffer + 1), SIZE_1 * sizeof(float));
#undef SIZE_1

  // CHECK: d_A = (float *)sycl::malloc_shared(size, q_ct1);
  hipMallocManaged((void **)&d_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(d_A = (float *)sycl::malloc_shared(size, q_ct1));
  errorCode = hipMallocManaged((void **)&d_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(d_A = (float *)sycl::malloc_shared(size, q_ct1)));
  MY_SAFE_CALL(hipMallocManaged((void **)&d_A, size));

  // CHECK: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) + size + sizeof(sycl::uchar4), q_ct1);
  // CHECK-NEXT: d_A = (float *)sycl::malloc_shared(sizeof(sycl::double2) * size * sizeof(sycl::uchar4), q_ct1);
  hipMallocManaged((void **)&d_A, sizeof(double2) + size + sizeof(uchar4));
  hipMallocManaged((void **)&d_A, sizeof(double2) * size * sizeof(uchar4));

  hipDeviceptr_t* D_ptr;
  // CHECK: *D_ptr = (dpct::device_ptr)sycl::malloc_shared(size, q_ct1);
  hipMallocManaged(D_ptr, size, hipMemAttachHost);

  /// memcpy

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode  = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  errorCode  = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  MY_SAFE_CALL(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
#define MACRO_A(x) size
#define MACRO_A2(x) MACRO_A(x)
#define MACRO_B size
#define MACOR_C(x) hipMemcpyDeviceToHost
#define MY_SAFE_CALL2(x) MY_SAFE_CALL(x)
  //CHECK: MY_SAFE_CALL2(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  MY_SAFE_CALL2(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));
  //CHECK: MY_SAFE_CALL2(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, MACRO_B)));
  MY_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_B, hipMemcpyDeviceToHost));
  //CHECK: MY_SAFE_CALL2(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, MACRO_A2(1))));
  MY_SAFE_CALL2(hipMemcpy(d_A, h_A, MACRO_A2(1), MACOR_C(1)));
  //CHECK: MY_SAFE_CALL2(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, foo_b(1))));
  MY_SAFE_CALL2(hipMemcpy(d_A, h_A, foo_b(1), MACOR_C(1)));

#define SIZE 100
  // CHECK: q_ct1.memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );

  /// memcpy async

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy(d_A, h_A, size);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memcpy(d_A, h_A, size));
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memcpy(d_A, h_A, size)));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream));

  // CHECK: dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3D(&parms);

  struct hipMemcpy3DParms *parms_pointer;
  // Followed call can't be processed.
  hipMemcpy3D(parms_pointer);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device, *stream);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host, *stream);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms, 0);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1, *stream);
  hipMemcpy3DAsync(&parms, stream);
  /// memcpy from symbol

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size).wait()));
  MY_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  /// memcpy from symbol async

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait()));
  MY_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  /// memcpy to symbol async

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));

  /// memset

  // CHECK: q_ct1.memset(d_A, 23, size).wait();
  hipMemset(d_A, 23, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size).wait());
  errorCode = hipMemset(d_A, 23, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size).wait()));
  MY_SAFE_CALL(hipMemset(d_A, 23, size));

  /// memset async

  // CHECK: q_ct1.memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size));
  errorCode = hipMemsetAsync(d_A, 23, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size)));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size));

  // CHECK: q_ct1.memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size));
  errorCode = hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size)));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, 0));

  // CHECK: stream->memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memset(d_A, 23, size));
  errorCode = hipMemsetAsync(d_A, 23, size, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memset(d_A, 23, size)));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, stream));

  // CHECK: dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK: dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);

  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size, *stream);
  hipMemset2DAsync(d_A, size, 0xf, size, size, stream);

  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e, 0);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e, *stream);
  hipMemset3DAsync(p_A, 0xf, e, stream);

  // CHECK: sycl::free(h_A, q_ct1);
  hipHostFree(h_A);
  // CHECK: errorCode = DPCT_CHECK_ERROR(sycl::free(h_A, q_ct1));
  errorCode = hipHostFree(h_A);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(sycl::free(h_A, q_ct1)));
  MY_SAFE_CALL(hipHostFree(h_A));

  // CHECK: *(void **)&d_A = (float *)h_A;
  hipHostGetDevicePointer((void **)&d_A, h_A, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(d_A = (float *)h_A);
  errorCode = hipHostGetDevicePointer(&d_A, h_A, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(d_A = (float *)h_A));
  MY_SAFE_CALL(hipHostGetDevicePointer(&d_A, h_A, 0));

  // CHECK: *D_ptr = (dpct::device_ptr)h_A;
  hipHostGetDevicePointer(D_ptr, h_A, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(*D_ptr = (dpct::device_ptr)h_A);
  errorCode = hipHostGetDevicePointer(D_ptr, h_A, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(*D_ptr = (dpct::device_ptr)h_A));
  MY_SAFE_CALL(hipHostGetDevicePointer(D_ptr, h_A, 0));

  hipHostRegister(h_A, size, 0);
  // CHECK: errorCode = 0;
  errorCode = hipHostRegister(h_A, size, 0);
  // CHECK: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipHostRegister(h_A, size, 0));

  hipHostUnregister(h_A);
  // CHECK: errorCode = 0;
  errorCode = hipHostUnregister(h_A);
  // CHECK: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipHostUnregister(h_A));
}


template <typename T>
int foo2() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;

  hipStream_t stream;
  /// memcpy from symbol

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, constData.get_ptr(), size));
  errorCode = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, constData.get_ptr(), size).wait()));
  MY_SAFE_CALL(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  /// memcpy from symbol async

  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: q_ct1.memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, constData.get_ptr(), size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, constData.get_ptr(), size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 1, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 2, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0));

  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size));
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size)));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream));

  /// memcpy to symbol
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), h_A, size));
  errorCode = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), h_A, size).wait()));
  MY_SAFE_CALL(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size));

  /// memcpy to symbol async
  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: q_ct1.memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 2, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0));

  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: errorCode = DPCT_CHECK_ERROR(stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size));
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(stream->memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size)));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream));
}

template int foo2<float>();
template int foo2<int>();

void foo3() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  hipArray_t a1;
  int deviceID = 0;

  // CHECK: auto s1 = std::make_shared<dpct::queue_ptr>((dpct::queue_ptr)&q_ct1);
  // CHECK: auto s2 = std::make_shared<dpct::queue_ptr>(&q_ct1);
  // CHECK: auto s3 = std::make_shared<dpct::queue_ptr>(&q_ct1);
  auto s1 = std::make_shared<hipStream_t>((hipStream_t)hipStreamDefault);
  auto s2 = std::make_shared<hipStream_t>(hipStreamLegacy);
  auto s3 = std::make_shared<hipStream_t>(hipStreamPerThread);

  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: q_ct1.memcpy(d_A, h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(d_A, h_A, size)));
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread));


  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy((char *)(constData.get_ptr()) + 1, h_A, size)));
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memcpy(h_A, (char *)(constData.get_ptr()) + 3, size)));
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device)));
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1)));
  hipMemcpy3DAsync(&parms, hipStreamDefault);
  hipMemcpy3DAsync(&parms, hipStreamLegacy);
  hipMemcpy3DAsync(&parms, hipStreamPerThread);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamDefault);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamLegacy);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: q_ct1.memset(d_A, 23, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(q_ct1.memset(d_A, 23, size)));
  hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(d_A, size, 0xf, size, size)));
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::async_dpct_memset(p_A, 0xf, e)));
  hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread));


  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100);
  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100);
  // CHECK: dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(deviceID).in_order_queue().prefetch(d_A,100)));
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault);
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy);
  hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy);
  errorCode = hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread);
  MY_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamDefault));
  MY_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamLegacy));
  MY_SAFE_CALL(hipMemPrefetchAsync (d_A, 100, deviceID, hipStreamPerThread));
  // CHECK: int cudevice = 0;
  hipDevice_t cudevice = 0;
  // CHECK: dpct::device_ptr devPtr;
  hipDeviceptr_t devPtr;
  // CHECK: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100);
  // CHECK: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100);
  // CHECK: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100);
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
  // CHECK: errorCode = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
  hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
  hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
  hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
  errorCode = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
  errorCode = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
  errorCode = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread));
}

/// cuda driver memory api
void foo4(){
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);

  int errorCode;
  // CHECK: /*
  // CHECK: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK: */
  // CHECK: h_A = (float *)sycl::malloc_host(size, q_ct1);
  hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: /*
  // CHECK: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK: */
  // CHECK: errorCode = DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1));
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: /*
  // CHECK: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK: */
  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(h_A = (float *)sycl::malloc_host(size, q_ct1)));
  MY_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocPortable));
  // CHECK: /*
  // CHECK: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK: */
  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::double2) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(double2) - size, hipHostMallocPortable);
  // CHECK: /*
  // CHECK: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the migrated code and was removed or replaced with 0. You may need to check the migrated code.
  // CHECK: */
  // CHECK: h_A = (float *)sycl::malloc_host(sizeof(sycl::uchar4) - size, q_ct1);
  hipHostAlloc((void **)&h_A, sizeof(uchar4) - size, hipHostMallocPortable);
}

#define MY_SAFE_CALL3(CALL) {                                               \
  hipError_t Error = CALL;                                                   \
  if (Error != hipSuccess) {                                               \
    printf("%s\n", hipGetErrorString(Error));                              \
    exit(Error);                                                            \
  }                                                                         \
}

void foo5(float* a) {
// CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
// CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
// CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
}


void foo6(float* a) {
  // CHECK: printf("%d\n", DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
  // CHECK: printf("%d\n", DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
  printf("%d\n", hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  printf("%d\n", hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
}

__global__ void test_kernel() {}

int foo7() {
  unsigned int mem_size;
  unsigned int *h_out_data;
  unsigned int *h_data;
  unsigned int *d_out_data;
  unsigned int *d_in_data_1;
  unsigned int *d_in_data_2;
  int num_data;

  for (unsigned int i = 0; i < num_data; i++)
    h_data[i] = i;
  // CHECK: q_ct1.memcpy(d_in_data_1, h_data, mem_size).wait();
  hipMemcpy(d_in_data_1, h_data, mem_size, hipMemcpyHostToDevice);

  for (unsigned int i = 0; i < num_data; i++)
    h_data[i] = num_data - 1 - i;
  // CHECK: q_ct1.memcpy(d_in_data_2, h_data, mem_size);
  hipMemcpy(d_in_data_2, h_data, mem_size, hipMemcpyHostToDevice);

  test_kernel<<<3, 3>>>();
  hipDeviceSynchronize();
  // CHECK: q_ct1.memcpy(h_out_data, d_out_data, mem_size).wait();
  hipMemcpy(h_out_data, d_out_data, mem_size, hipMemcpyDeviceToHost);

  return 0;
}

int foo8() {
  unsigned int mem_size;
  unsigned int *h_data;
  unsigned int *d_in_data_1;
  unsigned int *d_in_data_2;

  // CHECK: q_ct1.memcpy(d_in_data_1, h_data, mem_size);
  hipMemcpy(d_in_data_1, h_data, mem_size, hipMemcpyHostToDevice);
  // CHECK: q_ct1.memcpy(d_in_data_2, h_data, mem_size).wait();
  hipMemcpy(d_in_data_2, h_data, mem_size, hipMemcpyHostToDevice);
  return 0;
}

int foo9() {
  unsigned int mem_size;
  unsigned int *h_data;
  unsigned int *d_in_data_1;
  unsigned int *d_in_data_2;
  unsigned int *test = d_in_data_1;

  // CHECK: q_ct1.memcpy(d_in_data_1, h_data, mem_size).wait();
  hipMemcpy(d_in_data_1, h_data, mem_size, hipMemcpyHostToDevice);
  test;
  // CHECK: q_ct1.memcpy(d_in_data_2, h_data, mem_size).wait();
  hipMemcpy(d_in_data_2, h_data, mem_size, hipMemcpyHostToDevice);
  return 0;
}

int foo10(unsigned int *test) {
  unsigned int mem_size;
  unsigned int *data_d, *data_h;

  // CHECK: q_ct1.memcpy(data_d, data_h, mem_size).wait();
  hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
  test;
  // CHECK: q_ct1.memcpy(data_d, data_h, mem_size).wait();
  hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
  return 0;
}

unsigned int *global_test;

int foo11() {
  unsigned int mem_size;
  unsigned int *data_d, *data_h;

  // CHECK: q_ct1.memcpy(data_d, data_h, mem_size).wait();
  hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
  global_test;
  // CHECK: q_ct1.memcpy(data_d, data_h, mem_size).wait();
  hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
  return 0;
}

struct TEST {
  unsigned int t;
  void call() {
    unsigned int mem_size;
    unsigned int *data_d, *data_h;
    // CHECK: q_ct1.memcpy(data_d, data_h, mem_size);
    hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
    // CHECK: q_ct1.memcpy(data_d, data_h, mem_size).wait();
    hipMemcpy(data_d, data_h, mem_size, hipMemcpyHostToDevice);
  }
};

int foo12() {
  TEST test;
  return 0;
}

void foo13(float* a, bool flag) {
  // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
  // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  while(flag) {
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
    if(flag) {
      // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
      // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), a, 16).wait()));
      MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
      MY_SAFE_CALL3(hipMemcpyToSymbol(HIP_SYMBOL(constData), a, 16));
    } else {
      // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(constData.get_ptr(), a, 16)));
      // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
      MY_SAFE_CALL3(hipMemcpyToSymbol(HIP_SYMBOL(constData), a, 16));
      MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
    }
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  }

  do {
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, constData.get_ptr(), 16).wait()));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
    MY_SAFE_CALL3(hipMemcpyFromSymbol(a, HIP_SYMBOL(constData), 16));
  } while(flag);

  for(;;) {
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, constData.get_ptr(), 16)));
    // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
    MY_SAFE_CALL3(hipMemcpyFromSymbol(a, HIP_SYMBOL(constData), 16));
    MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  }
  // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16)));
  // CHECK: MY_SAFE_CALL3(DPCT_CHECK_ERROR(q_ct1.memcpy(a, a, 16).wait()));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
  MY_SAFE_CALL3(hipMemcpy(a, a, 16, hipMemcpyDeviceToHost));
}

void foo14() {
  int h_selected_num;
  int *d_selected_num;
  int *h_out;
  int *d_out;
  //CHECK:q_ct1.memcpy((void *)&h_selected_num, (void *)d_selected_num, sizeof(int)).wait();
  //CHECK-NEXT:q_ct1.memcpy((void *)h_out, (void *)d_out, h_selected_num * sizeof(int)).wait();
  hipMemcpy((void *)&h_selected_num, (void *)d_selected_num, sizeof(int), hipMemcpyDeviceToHost);
  hipMemcpy((void *)h_out, (void *)d_out, h_selected_num * sizeof(int), hipMemcpyDeviceToHost);
}

struct TEST_STR {
  int a[10];
};

void foo15() {
  std::vector<volatile TEST_STR *> buf;
  for (int i = 0; i < 32; i++) {
    //CHECK: buf[i] = (volatile TEST_STR *)sycl::malloc_host(sizeof(TEST_STR), dpct::get_in_order_queue());
    hipHostMalloc(&buf[i], sizeof(TEST_STR));
  }
}

void foo16() {
  std::vector<volatile TEST_STR *> buf;
  for (int i = 0; i < 32; i++) {
    //CHECK: (buf.front()) = (volatile TEST_STR *)sycl::malloc_host(sizeof(TEST_STR), dpct::get_in_order_queue());
    hipHostMalloc(&buf.front(), sizeof(TEST_STR));
  }
}

int foo17() {
  unsigned int mem_size;
  unsigned int *h_data;
  unsigned int *d_in_data_1;
  unsigned int *d_in_data_2;

  // CHECK: q_ct1.memcpy(d_in_data_1, h_data, mem_size).wait();
  hipMemcpy(d_in_data_1, h_data, mem_size, hipMemcpyHostToDevice);
  h_data[0] = 1;
  // CHECK: q_ct1.memcpy(d_in_data_2, h_data, mem_size).wait();
  hipMemcpy(d_in_data_2, h_data, mem_size, hipMemcpyHostToDevice);
  return 0;
}
