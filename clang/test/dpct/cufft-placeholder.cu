// RUN: dpct --format-range=none -out-root %T/cufft-placeholder %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-placeholder/cufft-placeholder.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
int rank;

//CHECK:void foo1(dpct::fft::fft_engine_ptr plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  plan->compute<sycl::double2, double>(idata, odata, dpct::fft::fft_direction::backward);
//CHECK-NEXT:}
void foo1(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

//CHECK:void foo2(dpct::fft::fft_engine_ptr plan) {
//CHECK-NEXT:  float* odata;
//CHECK-NEXT:  sycl::float2* idata;
//CHECK-NEXT:  plan->compute<sycl::float2, float>(idata, odata, dpct::fft::fft_direction::backward);
//CHECK-NEXT:}
void foo2(hipfftHandle plan) {
  float* odata;
  float2* idata;
  hipfftExecC2R(plan, idata, odata);
}

int main() {
  //CHECK:dpct::fft::fft_engine_ptr plan1;
  //CHECK-NEXT:dpct::fft::fft_type type1 = dpct::fft::fft_type::complex_double_to_real_double;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is aligning with the related "compute" function call.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan1->commit(&q_ct1, rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11, work_size);
  hipfftHandle plan1;
  hipfftType_t type1 = HIPFFT_Z2D;
  hipfftMakePlanMany(plan1, rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11, work_size);

  //CHECK:dpct::fft::fft_engine_ptr plan2;
  //CHECK-NEXT:dpct::fft::fft_type type2 = dpct::fft::fft_type::complex_float_to_real_float;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is aligning with the related "compute" function call.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan2->commit(&q_ct1, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);
  hipfftHandle plan2;
  hipfftType_t type2 = HIPFFT_C2R;
  hipfftMakePlanMany(plan2, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);

  foo1(plan1);
  foo2(plan2);

  return 0;
}

