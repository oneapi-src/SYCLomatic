// RUN: dpct --format-range=none -out-root %T/cufft-placeholder %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cufft-placeholder/cufft-placeholder.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


size_t* work_size;
int odist;
int ostride;
int * onembed;
int idist;
int istride;
int* inembed;
int * n;
int rank;

//CHECK:void foo1(std::shared_ptr<dpct::fft::fft_solver> plan) {
//CHECK-NEXT:  double* odata;
//CHECK-NEXT:  sycl::double2* idata;
//CHECK-NEXT:  plan->compute(idata, odata, dpct::fft::fft_dir::backward);
//CHECK-NEXT:}
void foo1(hipfftHandle plan) {
  double* odata;
  double2* idata;
  hipfftExecZ2D(plan, idata, odata);
}

//CHECK:void foo2(std::shared_ptr<dpct::fft::fft_solver> plan) {
//CHECK-NEXT:  float* odata;
//CHECK-NEXT:  sycl::float2* idata;
//CHECK-NEXT:  plan->compute(idata, odata, dpct::fft::fft_dir::backward);
//CHECK-NEXT:}
void foo2(hipfftHandle plan) {
  float* odata;
  float2* idata;
  hipfftExecC2R(plan, idata, odata);
}

int main() {
  //CHECK:std::shared_ptr<dpct::fft::fft_solver> plan1;
  //CHECK-NEXT:dpct::fft::fft_type type1 = dpct::fft::fft_type::complex_double_to_real_double;
  //CHECK-NEXT:plan1 = std::make_shared<dpct::fft::fft_solver>(rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11);
  hipfftHandle plan1;
  hipfftType_t type1 = HIPFFT_Z2D;
  hipfftMakePlanMany(plan1, rank, n, inembed, istride, idist, onembed, ostride, odist, type1, 11, work_size);

  //CHECK:std::shared_ptr<dpct::fft::fft_solver> plan2;
  //CHECK-NEXT:dpct::fft::fft_type type2 = dpct::fft::fft_type::complex_float_to_real_float;
  //CHECK-NEXT:plan2 = std::make_shared<dpct::fft::fft_solver>(rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12);
  hipfftHandle plan2;
  hipfftType_t type2 = HIPFFT_C2R;
  hipfftMakePlanMany(plan2, rank, n, inembed, istride, idist, onembed, ostride, odist, type2, 12, work_size);

  foo1(plan1);
  foo2(plan2);

  return 0;
}

