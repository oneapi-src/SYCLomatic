// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct -out-root %T/thrust_merge_by_key_usm %s --cuda-include-path="%cuda-path/include" --usm-level=restricted
// RUN: FileCheck --input-file %T/thrust_merge_by_key_usm/thrust_merge_by_key_usm.dp.cpp --match-full-lines %s

#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

int main(void) {

  thrust::device_vector<int> AD(4);
  thrust::device_vector<int> BD(4);
  thrust::device_vector<int> CD(4);  
  thrust::device_vector<int> DD(4);
  thrust::device_vector<int> ED(8);
  thrust::device_vector<int> FD(8);
  
  thrust::host_vector<int> AH(4);
  thrust::host_vector<int> BH(4);
  thrust::host_vector<int> CH(4);  
  thrust::host_vector<int> DH(4);
  thrust::host_vector<int> EH(8);
  thrust::host_vector<int> FH(8);    
  
  int *h_ptr;
  int *d_ptr;

  h_ptr = (int*)std::malloc(20 * sizeof(int));
  hipMalloc(&d_ptr, 20 * sizeof(int));

/*******************************************************************************************
 1. Test merge_by_key
 2. Test four VERSIONs (with/without exec argument) AND (with/without comparator)
 3. Test each VERSION with (device_vector/host_vector/malloc-ed memory/hipMalloc-ed memory)
 *******************************************************************************************/

/*********** merge_by_key ***********************************************************************************************************************************************/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::merge(oneapi::dpl::execution::par_unseq, AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::par_unseq, h_ptr, h_ptr + 4, BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
  // VERSION                           first1      last1     first2      last2     val1        val2        keys        values
  thrust::merge_by_key(                AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
  thrust::merge_by_key(                AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());
  thrust::merge_by_key(                h_ptr,      h_ptr+4,  BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
  // Overload not supported with thrust
  // thrust::merge_by_key(                d_ptr,      d_ptr+4,  BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::merge(oneapi::dpl::execution::par_unseq, AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), std::greater<int>());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), std::greater<int>());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::par_unseq, h_ptr, h_ptr + 4, BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), std::greater<int>());
  // VERSION                           first1      last1     first2      last2     val1        val2        keys        values      comparator
  thrust::merge_by_key(                AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), thrust::greater<int>());
  thrust::merge_by_key(                AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), thrust::greater<int>());
  thrust::merge_by_key(                h_ptr,      h_ptr+4,  BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), thrust::greater<int>());  
#ifdef ADD_BUG
  // This fails with nvcc
  thrust::merge_by_key(                d_ptr,      d_ptr+4,  BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), thrust::greater<int>());
#endif

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::merge(oneapi::dpl::execution::par_unseq, AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::par_unseq, h_ptr, h_ptr + 4, BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), d_ptr, d_ptr + 4, BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());
  // VERSION                           first1      last1     first2      last2     val1        val2        keys        values
  thrust::merge_by_key(thrust::host,   AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
  thrust::merge_by_key(thrust::device, AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());
  thrust::merge_by_key(thrust::host,   h_ptr,      h_ptr+4,  BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin());
  thrust::merge_by_key(thrust::device, d_ptr,      d_ptr+4,  BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin());

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::merge(oneapi::dpl::execution::par_unseq, AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), std::greater<int>());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), std::greater<int>());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::par_unseq, h_ptr, h_ptr + 4, BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), std::greater<int>());
// CHECK-NEXT:dpct::merge(oneapi::dpl::execution::make_device_policy(q_ct1), d_ptr, d_ptr + 4, BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), std::greater<int>());
  // VERSION                           first1      last1     first2      last2     val1        val2        keys        values      comparator
  thrust::merge_by_key(thrust::host,   AH.begin(), AH.end(), BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), thrust::greater<int>());
  thrust::merge_by_key(thrust::device, AD.begin(), AD.end(), BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), thrust::greater<int>());
  thrust::merge_by_key(thrust::host,   h_ptr,      h_ptr+4,  BH.begin(), BH.end(), CH.begin(), DH.begin(), EH.begin(), FH.begin(), thrust::greater<int>());
  thrust::merge_by_key(thrust::device, d_ptr,      d_ptr+4,  BD.begin(), BD.end(), CD.begin(), DD.begin(), ED.begin(), FD.begin(), thrust::greater<int>());

  return 0;
}
