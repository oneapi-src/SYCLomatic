
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/conflict-resolution %s -passes "ErrorHandlingIfStmtRule,ErrorConstantsRule" --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/conflict-resolution/conflict-resolution.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/conflict-resolution/conflict-resolution.dp.cpp -o %T/conflict-resolution/conflict-resolution.dp.o %}

int printf(const char *format, ...);

// CHECK: void test_00(cudaError_t err) {
// CHECK-NEXT:   {{ +}}
// CHECK-NEXT: }
void test_00(hipError_t err) {
  if (err != hipSuccess) {
    printf("Some error happenned\n");
    exit(1);
  }
}

