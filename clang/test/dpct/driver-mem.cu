// RUN: dpct --format-range=none -out-root %T/driver-mem %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem/driver-mem.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/driver-mem/driver-mem.dp.cpp -o %T/driver-mem/driver-mem.dp.o %}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    size_t count = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)sycl::malloc_host(size, q_ct1);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    hipDeviceptr_t p1;
    [&p1]() {
        //CHECK:p1 = 0;
        p1 = 0;
    }();

    hipDeviceptr_t p2;
    [&]() {
        //CHECK:p2 = 0;
        p2 = 0;
    }();

    // CHECK: dpct::device_ptr f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: dpct::device_ptr f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = (dpct::device_ptr)sycl::malloc_device(size, q_ct1);
    hipMalloc(&f_D, size);

    // CHECK: dpct::queue_ptr stream;
    hipStream_t stream;
    // CHECK: stream->memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: q_ct1.memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_A, size).wait());
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: stream->memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: q_ct1.memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: q_ct1.memcpy(f_A, f_D, size).wait();
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: r = DPCT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    hipMemcpyDtoD(f_D, f_D2, size);
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    cuMemcpy(f_D, f_D2, size);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size).wait());
    CALL(cuMemcpy(f_D, f_D2, size));
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = cuMemcpy(f_D, f_D2, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, stream);
    // CHECK: CALL(stream->memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, stream));
    // CHECK: r = DPCT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, 0));
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, 0);
    unsigned int v32 = 50000;
    unsigned short v16 = 20000;
    unsigned char v8 = (unsigned char) 200;
    //CHECK: dpct::dpct_memset_d32(f_D, v32, size);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, v16, size * 2);
    //CHECK-NEXT: dpct::dpct_memset(f_D, v8, size * 4);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, v32, size, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, v16, size * 2, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, v8, size * 4, *stream);
    //CHECK-NEXT: dpct::dpct_memset_d32(f_D, 1, v32, 4, 6);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, 1, v16, 4 * 2, 6);
    //CHECK-NEXT: dpct::dpct_memset(f_D, 1, v8, 4 * 4, 6);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, 1, v32, 4, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, 1, v16, 4 * 2, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, 1, v8, 4 * 4, 6, *stream);
    hipMemsetD32(f_D, v32, size);
    hipMemsetD16(f_D, v16, size * 2);
    hipMemsetD8(f_D, v8, size * 4);
    hipMemsetD32Async(f_D, v32, size, stream);
    hipMemsetD16Async(f_D, v16, size * 2, stream);
    hipMemsetD8Async(f_D, v8, size * 4, stream);
    hipMemsetD2D32(f_D, 1, v32, 4, 6);
    hipMemsetD2D16(f_D, 1, v16, 4 * 2, 6);
    hipMemsetD2D8(f_D, 1, v8, 4 * 4, 6);
    hipMemsetD2D32Async(f_D, 1, v32, 4, 6, stream);
    hipMemsetD2D16Async(f_D, 1, v16, 4 * 2, 6, stream);
    hipMemsetD2D8Async(f_D, 1, v8, 4 * 4, 6, stream);

    // CHECK: dpct::mem_cpy_param_wrapper cpy;
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy.set_ptr_to_data(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy.set_p_to_data(20);
    cpy.dstPitch = 20;
    // CHECK: cpy.set_y_to_pos(10);
    cpy.dstY = 10;
    // CHECK: cpy.set_x_to_pos(15);
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy.set_ptr_from_data(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy.set_p_from_data(20);
    cpy.srcPitch = 20;
    // CHECK: cpy.set_y_from_pos(10);
    cpy.srcY = 10;
    // CHECK: cpy.set_x_from_pos(15);
    cpy.srcXInBytes = 15;

    // CHECK: cpy.set_x(4);
    cpy.WidthInBytes = 4;
    // CHECK: cpy.set_y(7);
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(&cpy);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(&cpy, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;

    hipError_t cu_err;

    hipDeviceptr_t cuDevPtr;

    hipDevice_t cudevice;
    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, advise);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, advise)));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: cu_err = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, advise));
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0));
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::cpu_device().in_order_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, hipCpuDeviceId);


    hipDeviceptr_t devPtr;
    hipError_t curesult;
    // CHECK: stream->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, stream);
    // CHECK: (*&stream)->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, *&stream);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, NULL);
    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault));
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy));
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).in_order_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread));

    // CHECK: dpct::mem_cpy_param_wrapper cpy2;
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2.set_to_image_data(ca);
    cpy2.dstArray = ca;
    // CHECK: cpy2.set_p_to_data(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2.set_y_to_data(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2.set_y_to_pos(3);
    cpy2.dstY = 3;
    // CHECK: cpy2.set_z_to_pos(2);
    cpy2.dstZ = 2;
    // CHECK: cpy2.set_x_to_pos(1);
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2.set_ptr_from_data(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2.set_p_from_data(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2.set_y_from_data(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2.set_y_from_pos(3);
    cpy2.srcY = 3;
    // CHECK: cpy2.set_z_from_pos(2);
    cpy2.srcZ = 2;
    // CHECK: cpy2.set_x_from_pos(1);
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2.set_x(3);
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2.set_y(2);
    cpy2.Height = 2;
    // CHECK: cpy2.set_z(1);
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(&cpy2);
    hipDrvMemcpy3D(&cpy2);

    hipStream_t cs;
    // CHECK: dpct::async_dpct_memcpy(&cpy2, *cs);
    hipDrvMemcpy3DAsync(&cpy2, cs);

    float *h_A = (float *)malloc(100);
    // CHECK:sycl::free(h_A, q_ct1);
    hipHostFree(h_A);
    // CHECK:sycl::free(f_D, q_ct1);
    hipFree(f_D);

    unsigned int flags;
    int host;


    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(flags = 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
