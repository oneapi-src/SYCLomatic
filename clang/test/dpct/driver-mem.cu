// RUN: dpct --format-range=none -out-root %T/driver-mem %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem/driver-mem.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    size_t count = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)sycl::malloc_host(size, q_ct1);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    // CHECK: char * f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: char * f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = (char *)sycl::malloc_device(size, q_ct1);
    hipMalloc(&f_D, size);

    // CHECK: sycl::queue * stream;
    hipStream_t stream;
    // CHECK: stream->memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: q_ct1.memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_A, size).wait());
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: stream->memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: q_ct1.memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: q_ct1.memcpy(f_A, f_D, size).wait();
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: r = (stream->memcpy(f_D, f_D2, size), 0);
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: r = (q_ct1.memcpy(f_D, f_D2, size), 0);
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    hipMemcpyDtoD(f_D, f_D2, size);
    // CHECK: r = (q_ct1.memcpy(f_D, f_D2, size).wait(), 0);
    r = hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: dpct::pitched_data cpy_from_data_ct1, cpy_to_data_ct1;
    // CHECK: sycl::id<3> cpy_from_pos_ct1(0, 0, 0), cpy_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy_size_ct1(1, 1, 1);
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy_to_data_ct1.set_data_ptr(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy_to_data_ct1.set_pitch(20);
    cpy.dstPitch = 20;
    // CHECK: cpy_to_pos_ct1[1] = 10;
    cpy.dstY = 10;
    // CHECK: cpy_to_pos_ct1[0] = 15;
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy_from_data_ct1.set_data_ptr(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy_from_data_ct1.set_pitch(20);
    cpy.srcPitch = 20;
    // CHECK: cpy_from_pos_ct1[1] = 10;
    cpy.srcY = 10;
    // CHECK: cpy_from_pos_ct1[0] = 15;
    cpy.srcXInBytes = 15;

    // CHECK: cpy_size_ct1[0] = 4;
    cpy.WidthInBytes = 4;
    // CHECK: cpy_size_ct1[1] = 7;
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1, dpct::automatic, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;

    hipError_t cu_err;

    hipDeviceptr_t cuDevPtr;

    hipDevice_t cudevice;
    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise), 0));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise), 0);
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0), 0));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0), 0));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0), 0));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0), 0));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0), 0);
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::cpu_device().default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, hipCpuDeviceId);


    hipDeviceptr_t devPtr;
    hipError_t curesult;
    // CHECK: stream->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, stream);
    // CHECK: (*&stream)->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, *&stream);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: curesult = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0);
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, NULL);
    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: curesult = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0);
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: curesult = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0);
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: curesult = (dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0);
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault));
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy));
    // CHECK: /*
    // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError((dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100), 0));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread));

    // CHECK: dpct::pitched_data cpy2_from_data_ct1, cpy2_to_data_ct1;
    // CHECK: sycl::id<3> cpy2_from_pos_ct1(0, 0, 0), cpy2_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy2_size_ct1(1, 1, 1);
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2_to_data_ct1 = ca->to_pitched_data();
    cpy2.dstArray = ca;
    // CHECK: cpy2_to_data_ct1.set_pitch(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2_to_data_ct1.set_y(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2_to_pos_ct1[1] = 3;
    cpy2.dstY = 3;
    // CHECK: cpy2_to_pos_ct1[2] = 2;
    cpy2.dstZ = 2;
    // CHECK: cpy2_to_pos_ct1[0] = 1;
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2_from_data_ct1.set_data_ptr(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2_from_data_ct1.set_pitch(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2_from_data_ct1.set_y(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2_from_pos_ct1[1] = 3;
    cpy2.srcY = 3;
    // CHECK: cpy2_from_pos_ct1[2] = 2;
    cpy2.srcZ = 2;
    // CHECK: cpy2_from_pos_ct1[0] = 1;
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2_size_ct1[0] = 3;
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2_size_ct1[1] = 2;
    cpy2.Height = 2;
    // CHECK: cpy2_size_ct1[2] = 1;
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1);
    hipDrvMemcpy3D(&cpy2);

    float *h_A = (float *)malloc(100);
    // CHECK:sycl::free(h_A, q_ct1);
    hipHostFree(h_A);
    // CHECK:sycl::free(f_D, q_ct1);
    hipFree(f_D);

    unsigned int flags;
    int host;


    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError((flags = 0, 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
