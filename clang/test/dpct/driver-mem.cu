// RUN: dpct --format-range=none -out-root %T/driver-mem %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem/driver-mem.dp.cpp %s

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    size_t count = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)sycl::malloc_host(size, q_ct1);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    hipDeviceptr_t p1;
    [&p1]() {
        //CHECK:p1 = 0;
        p1 = 0;
    }();

    hipDeviceptr_t p2;
    [&]() {
        //CHECK:p2 = 0;
        p2 = 0;
    }();

    // CHECK: dpct::device_ptr f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: dpct::device_ptr f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: f_D = (dpct::device_ptr)sycl::malloc_device(size, q_ct1);
    hipMalloc(&f_D, size);

    // CHECK: dpct::queue_ptr stream;
    hipStream_t stream;
    // CHECK: stream->memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: q_ct1.memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_A, size).wait());
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: stream->memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: q_ct1.memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: q_ct1.memcpy(f_A, f_D, size).wait();
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: r = DPCT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    hipMemcpyDtoD(f_D, f_D2, size);
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    cuMemcpy(f_D, f_D2, size);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size).wait());
    CALL(cuMemcpy(f_D, f_D2, size));
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = cuMemcpy(f_D, f_D2, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, stream);
    // CHECK: CALL(stream->memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, stream));
    // CHECK: r = DPCT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, 0));
    // CHECK: r = DPCT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, 0);
    unsigned int v32 = 50000;
    unsigned short v16 = 20000;
    unsigned char v8 = (unsigned char) 200;
    //CHECK: dpct::dpct_memset_d32(f_D, v32, size);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, v16, size * 2);
    //CHECK-NEXT: dpct::dpct_memset(f_D, v8, size * 4);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, v32, size, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, v16, size * 2, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, v8, size * 4, *stream);
    //CHECK-NEXT: dpct::dpct_memset_d32(f_D, 1, v32, 4, 6);
    //CHECK-NEXT: dpct::dpct_memset_d16(f_D, 1, v16, 4 * 2, 6);
    //CHECK-NEXT: dpct::dpct_memset(f_D, 1, v8, 4 * 4, 6);
    //CHECK-NEXT: dpct::async_dpct_memset_d32(f_D, 1, v32, 4, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset_d16(f_D, 1, v16, 4 * 2, 6, *stream);
    //CHECK-NEXT: dpct::async_dpct_memset(f_D, 1, v8, 4 * 4, 6, *stream);
    hipMemsetD32(f_D, v32, size);
    hipMemsetD16(f_D, v16, size * 2);
    hipMemsetD8(f_D, v8, size * 4);
    hipMemsetD32Async(f_D, v32, size, stream);
    hipMemsetD16Async(f_D, v16, size * 2, stream);
    hipMemsetD8Async(f_D, v8, size * 4, stream);
    hipMemsetD2D32(f_D, 1, v32, 4, 6);
    hipMemsetD2D16(f_D, 1, v16, 4 * 2, 6);
    hipMemsetD2D8(f_D, 1, v8, 4 * 4, 6);
    hipMemsetD2D32Async(f_D, 1, v32, 4, 6, stream);
    hipMemsetD2D16Async(f_D, 1, v16, 4 * 2, 6, stream);
    hipMemsetD2D8Async(f_D, 1, v8, 4 * 4, 6, stream);

    // CHECK: dpct::pitched_data cpy_from_data_ct1, cpy_to_data_ct1;
    // CHECK: sycl::id<3> cpy_from_pos_ct1(0, 0, 0), cpy_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy_size_ct1(1, 1, 1);
    hip_Memcpy2D cpy;
    //
    cpy.dstMemoryType = hipMemoryTypeHost;
    // CHECK: cpy_to_data_ct1.set_data_ptr(f_A);
    cpy.dstHost = f_A;
    // CHECK: cpy_to_data_ct1.set_pitch(20);
    cpy.dstPitch = 20;
    // CHECK: cpy_to_pos_ct1[1] = 10;
    cpy.dstY = 10;
    // CHECK: cpy_to_pos_ct1[0] = 15;
    cpy.dstXInBytes = 15;

    //
    cpy.srcMemoryType = hipMemoryTypeDevice;
    // CHECK: cpy_from_data_ct1.set_data_ptr(f_D);
    cpy.srcDevice = f_D;
    // CHECK: cpy_from_data_ct1.set_pitch(20);
    cpy.srcPitch = 20;
    // CHECK: cpy_from_pos_ct1[1] = 10;
    cpy.srcY = 10;
    // CHECK: cpy_from_pos_ct1[0] = 15;
    cpy.srcXInBytes = 15;

    // CHECK: cpy_size_ct1[0] = 4;
    cpy.WidthInBytes = 4;
    // CHECK: cpy_size_ct1[1] = 7;
    cpy.Height = 7;

    // CHECK: dpct::dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1);
    hipMemcpyParam2D(&cpy);
    // CHECK: dpct::async_dpct_memcpy(cpy_to_data_ct1, cpy_to_pos_ct1, cpy_from_data_ct1, cpy_from_pos_ct1, cpy_size_ct1, dpct::automatic, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;

    hipError_t cu_err;

    hipDeviceptr_t cuDevPtr;

    hipDevice_t cudevice;
    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise)));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: cu_err = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, advise));
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0));
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::dev_mgr::instance().get_device(cudevice).default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: dpct::cpu_device().default_queue().mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, hipCpuDeviceId);


    hipDeviceptr_t devPtr;
    hipError_t curesult;
    // CHECK: stream->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, stream);
    // CHECK: (*&stream)->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, *&stream);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, NULL);
    // CHECK: dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
    // CHECK: curesult = DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault));
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy));
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(dpct::dev_mgr::instance().get_device(cudevice).default_queue().prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread));

    // CHECK: dpct::pitched_data cpy2_from_data_ct1, cpy2_to_data_ct1;
    // CHECK: sycl::id<3> cpy2_from_pos_ct1(0, 0, 0), cpy2_to_pos_ct1(0, 0, 0);
    // CHECK: sycl::range<3> cpy2_size_ct1(1, 1, 1);
    HIP_MEMCPY3D cpy2;

    hipArray_t ca;
    //
    cpy2.dstMemoryType = hipMemoryTypeArray;
    // CHECK: cpy2_to_data_ct1 = ca->to_pitched_data();
    cpy2.dstArray = ca;
    // CHECK: cpy2_to_data_ct1.set_pitch(5);
    cpy2.dstPitch = 5;
    // CHECK: cpy2_to_data_ct1.set_y(4);
    cpy2.dstHeight = 4;
    // CHECK: cpy2_to_pos_ct1[1] = 3;
    cpy2.dstY = 3;
    // CHECK: cpy2_to_pos_ct1[2] = 2;
    cpy2.dstZ = 2;
    // CHECK: cpy2_to_pos_ct1[0] = 1;
    cpy2.dstXInBytes = 1;
    //
    cpy2.dstLOD = 0;

    //
    cpy2.srcMemoryType = hipMemoryTypeHost;
    // CHECK: cpy2_from_data_ct1.set_data_ptr(f_A);
    cpy2.srcHost = f_A;
    // CHECK: cpy2_from_data_ct1.set_pitch(5);
    cpy2.srcPitch = 5;
    // CHECK: cpy2_from_data_ct1.set_y(4);
    cpy2.srcHeight = 4;
    // CHECK: cpy2_from_pos_ct1[1] = 3;
    cpy2.srcY = 3;
    // CHECK: cpy2_from_pos_ct1[2] = 2;
    cpy2.srcZ = 2;
    // CHECK: cpy2_from_pos_ct1[0] = 1;
    cpy2.srcXInBytes = 1;
    //
    cpy2.srcLOD = 0;

    // CHECK: cpy2_size_ct1[0] = 3;
    cpy2.WidthInBytes = 3;
    // CHECK: cpy2_size_ct1[1] = 2;
    cpy2.Height = 2;
    // CHECK: cpy2_size_ct1[2] = 1;
    cpy2.Depth = 1;

    // CHECK: dpct::dpct_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1);
    hipDrvMemcpy3D(&cpy2);

    hipStream_t cs;
    // CHECK: dpct::async_dpct_memcpy(cpy2_to_data_ct1, cpy2_to_pos_ct1, cpy2_from_data_ct1, cpy2_from_pos_ct1, cpy2_size_ct1, dpct::automatic, *cs);
    hipDrvMemcpy3DAsync(&cpy2, cs);

    float *h_A = (float *)malloc(100);
    // CHECK:sycl::free(h_A, q_ct1);
    hipHostFree(h_A);
    // CHECK:sycl::free(f_D, q_ct1);
    hipFree(f_D);

    unsigned int flags;
    int host;


    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError(DPCT_CHECK_ERROR(flags = 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
