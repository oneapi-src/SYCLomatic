// RUN: dpct -out-root %T/driver-stream-and-event %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-stream-and-event/driver-stream-and-event.dp.cpp %s


#include <hip/hip_runtime.h>
#include<vector>
// CHECK: #include <future>
template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

void foo(){
  hipFunction_t f;
  hipStream_t s;
  hipEvent_t e;

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuFuncSetCacheConfig was removed because DPC++ currently does not support configuring shared memory on devices.
  //CHECK-NEXT: */
  cuFuncSetCacheConfig(f, hipFuncCachePreferNone);

  //CHECK: s = dpct::get_current_device().create_queue();
  //CHECK-NEXT: s->wait();
  hipStreamCreateWithFlags(&s, hipStreamDefault);
  hipStreamSynchronize(s);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuEventCreate was removed because this call is redundant in DPC++.
  //CHECK-NEXT: */
  //CHECK-NEXT: e = s->ext_oneapi_submit_barrier({e});
  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

  //CHECK: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: e_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: e = s->ext_oneapi_submit_barrier();
  //CHECK-NEXT: e.wait_and_throw();
  hipEventRecord(e, s);
  hipEventSynchronize(e);

  //CHECK: sycl::info::event_command_status r;
  //CHECK-NEXT: r = e.get_info<sycl::info::event::command_execution_status>();
  hipError_t r;
  r = hipEventQuery(e);

  //CHECK: sycl::event start, end;
  //CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
  //CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> end_ct1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: start_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  //CHECK-NEXT: */
  //CHECK-NEXT: end_ct1 = std::chrono::steady_clock::now();
  //CHECK-NEXT: float result_time;
  //CHECK-NEXT: result_time = std::chrono::duration<float, std::milli>(end_ct1 - start_ct1).count();
  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);

  int rr;
  //CHECK: rr = dpct::get_kernel_function_info((const void *)f).max_work_group_size;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);

}


// CHECK: void process(sycl::queue *st, char *data, int status) {}
void process(hipStream_t st, char *data, hipError_t status) {}

template<typename T>
// CHECK: void callback(sycl::queue *hStream, int status, void *userData) {
void callback(hipStream_t hStream, hipError_t status, void* userData) {
  T *data = static_cast<T *>(userData);
  process(hStream, data, status);
}

void test_stream() {
  hipStream_t hStream;
  void* data;
  unsigned int flag;
  size_t length;
  hipDeviceptr_t  cuPtr;
  // CHECK: std::async([&]() {hStream->wait(); callback<char>(hStream, 0, data); });
  hipStreamAddCallback(hStream, callback<char>, data, flag);

  // CHECK: int result = (std::async([&]() {hStream->wait(); callback<char>(hStream, 0, data); }), 0);
  hipError_t result = hipStreamAddCallback(hStream, callback<char>, data, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuStreamAttachMemAsync was removed because DPC++ currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because DPC++ currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: result = 0;
  result = hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because DPC++ currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: MY_ERROR_CHECKER(0);
  MY_ERROR_CHECKER(hipStreamAttachMemAsync(hStream, cuPtr, length, flag));
  hipStreamAttachMemAsync(hStream, cuPtr, std::vector<int>(1,1).front(), flag);
  // CHECK: dpct::get_current_device().destroy_queue(hStream);
  hipStreamDestroy(hStream);
}
