// RUN: dpct -out-root %T/driver-stream-and-event %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-stream-and-event/driver-stream-and-event.dp.cpp %s


#include <hip/hip_runtime.h>
#include<vector>
// CHECK: #include <future>
template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

void foo(){
  hipFunction_t f;
  hipStream_t s;
  hipEvent_t e;

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuFuncSetCacheConfig was removed because SYCL currently does not support configuring shared memory on devices.
  //CHECK-NEXT: */
  cuFuncSetCacheConfig(f, hipFuncCachePreferNone);

  //CHECK: s = dpct::get_current_device().create_queue();
  //CHECK-NEXT: s->wait();
  hipStreamCreateWithFlags(&s, hipStreamDefault);
  hipStreamSynchronize(s);

  //CHECK: s->ext_oneapi_submit_barrier({*e});
  hipEventCreateWithFlags(&e, hipEventDefault);
  hipStreamWaitEvent(s, e, 0);

  //CHECK: *e = s->ext_oneapi_submit_barrier();
  //CHECK-NEXT: e->wait_and_throw();
  hipEventRecord(e, s);
  hipEventSynchronize(e);

  //CHECK: sycl::info::event_command_status r;
  //CHECK-NEXT: r = e->get_info<sycl::info::event::command_execution_status>();
  hipError_t r;
  r = hipEventQuery(e);

  //CHECK: dpct::event_ptr start, end;
  //CHECK: *start = s->ext_oneapi_submit_barrier();
  //CHECK: *end = s->ext_oneapi_submit_barrier();
  //CHECK: start->wait_and_throw();
  //CHECK: end->wait_and_throw();
  //CHECK: float result_time;
  //CHECK: result_time = (end->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
  hipEvent_t start, end;
  hipEventRecord(start, s);
  hipEventRecord(end, s);
  hipEventSynchronize(start);
  hipEventSynchronize(end);
  float result_time;
  hipEventElapsedTime(&result_time, start, end);

  int rr;
  //CHECK: rr = dpct::get_kernel_function_info((const void *)f).max_work_group_size;
  hipFuncGetAttribute(&rr, HIP_FUNC_ATTRIBUTE_MAX_THREADS_PER_BLOCK, f);

  hipEventDestroy(start);

  hipEventDestroy(end);
}


// CHECK: void process(dpct::queue_ptr st, char *data, int status) {}
void process(hipStream_t st, char *data, hipError_t status) {}

template<typename T>
// CHECK: void callback(dpct::queue_ptr hStream, int status, void *userData) {
void callback(hipStream_t hStream, hipError_t status, void* userData) {
  T *data = static_cast<T *>(userData);
  process(hStream, data, status);
}

void test_stream() {
  hipStream_t hStream;
  void* data;
  unsigned int flag;
  size_t length;
  hipDeviceptr_t  cuPtr;
  // CHECK: std::async([&]() {hStream->wait(); callback<char>(hStream, 0, data); });
  hipStreamAddCallback(hStream, callback<char>, data, flag);

  // CHECK: int result = (std::async([&]() {hStream->wait(); callback<char>(hStream, 0, data); }), 0);
  hipError_t result = hipStreamAddCallback(hStream, callback<char>, data, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuStreamAttachMemAsync was removed because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: result = 0;
  result = hipStreamAttachMemAsync(hStream, cuPtr, length, flag);

  //CHECK: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cuStreamAttachMemAsync was replaced with 0 because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  //CHECK-NEXT: MY_ERROR_CHECKER(0);
  MY_ERROR_CHECKER(hipStreamAttachMemAsync(hStream, cuPtr, length, flag));

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cuStreamAttachMemAsync was removed because SYCL currently does not support associating USM with a specific queue.
  //CHECK-NEXT: */
  hipStreamAttachMemAsync(hStream, cuPtr, std::vector<int>(1,1).front(), flag);

  // CHECK: dpct::get_current_device().destroy_queue(hStream);
  hipStreamDestroy(hStream);
}
