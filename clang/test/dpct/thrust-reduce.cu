#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/thrust-reduce %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -fno-delayed-template-parsing -std=c++17 -fsized-deallocation -ferror-limit=50
// RUN: FileCheck --input-file %T/thrust-reduce/thrust-reduce.dp.cpp --match-full-lines %s
// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT: #include <oneapi/dpl/algorithm>
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <dpct/dpl_utils.hpp>
#include <thrust/device_ptr.h>
#include <thrust/reduce.h>
#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/transform_reduce.h>

template <typename T>
struct square {
// CHECK:  T operator()(const T& x) const { return x * x; }
  __host__ __device__  T operator()(const T& x) const { return x * x; }
};

int main() {
  double sum;
  double *p;
// CHECK:  dpct::device_pointer<double> dp(p);
  thrust::device_ptr<double> dp(p);
// CHECK:  sum = std::reduce(oneapi::dpl::execution::make_device_policy(dpct::get_default_queue()), dp, dp + 10);
  sum = thrust::reduce(dp, dp + 10);
}

void check_transform_reduce() {
// CHECK:  dpct::device_ext &dev_ct1 = dpct::get_current_device();
// CHECK-NEXT:  sycl::queue &q_ct1 = dev_ct1.default_queue();
  float x[4] = {1.0, 2.0, 3.0, 4.0};
// CHECK:  dpct::device_vector<float> d_x(x, x + 4);
  thrust::device_vector<float> d_x(x, x + 4);
  square<float>        unary_op;
// CHECK:  std::plus<float> binary_op;
  thrust::plus<float> binary_op;
  float init = 0;

// CHECK:  float norm     = std::transform_reduce(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), init, binary_op, unary_op);
  float norm     = thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op);
// CHECK:  float normSqrt = std::sqrt(std::transform_reduce(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), init, binary_op, unary_op));
  float normSqrt = std::sqrt(thrust::transform_reduce(d_x.begin(), d_x.end(), unary_op, init, binary_op));
}

template <typename T>
class C {
  T *data;
public:
  C() {
    this->data = 0;
  }

  // CHECK:   inline T *raw() {
  // CHECK-NEXT:   return dpct::get_raw_pointer(this->data);
  // CHECK-NEXT: }
  // CHECK-NEXT: inline const T *raw() const {
  // CHECK-NEXT:   return dpct::get_raw_pointer(this->data + 2);
  // CHECK-NEXT: }
  inline T *raw() {
    return thrust::raw_pointer_cast(this->data);
  }
  inline const T *raw() const {
    return thrust::raw_pointer_cast(this->data + 2);
  }
};

