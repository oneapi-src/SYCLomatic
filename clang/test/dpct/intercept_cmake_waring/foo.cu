// UNSUPPORTED: system-windows
// intercept-build only supports Linux
//
// ------ prepare test directory
// RUN: cd %T
// RUN: cp %s foo.cu
// RUN: cp %S/Makefile ./Makefile
// RUN: cp %S/CMakeLists.txt ./CMakeLists.txt
// RUN: cp %S/bar.cpp bar.cpp
// RUN: intercept-build make > intercept_log.txt 2>&1
// RUN: grep "cmake is called to generate project build" ./intercept_log.txt

#include <hip/hip_runtime.h>
#include <iostream>

int main() {
  return 0;
}
