// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/normback %S/normback.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/normback/normback.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/normback/normback.dp.cpp -o %T/normback/normback.dp.o %}
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 1, sbw = 1;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    float *diffout, *diffdata, *diffscale, *diffbias, *diffz;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(oele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);
    std::vector<float> host_diffout(oele_num, 0.f);
    std::vector<float> host_diffz(oele_num, 0.f);
    std::vector<float> host_diffdata(ele_num, 0.f);
    std::vector<float> host_diffscale(sele_num, 1.0f);
    std::vector<float> host_diffbias(sele_num, 0.0f);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
        host_z[i] = 10;
        host_diffout[i] = 100 * i;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * oele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);
    hipMalloc(&diffout, sizeof(float) * oele_num);
    hipMalloc(&diffz, sizeof(float) * oele_num);
    hipMalloc(&diffdata, sizeof(float) * ele_num);
    hipMalloc(&diffscale, sizeof(float) * sele_num);
    hipMalloc(&diffbias, sizeof(float) * sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), sizeof(float) * oele_num,
      hipMemcpyHostToDevice);
    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.5f;
    // CHECK: dpct::dnnl::activation_desc ActivationDesc;
    // CHECK: ActivationDesc.set(dnnl::algorithm::eltwise_relu_use_dst_for_bwd, 0.0f);
    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);

    float *workspace, *reservespace;
    size_t workspace_size, reservespace_size;
    // CHECK: workspace_size = 0;
    cudnnGetNormalizationForwardTrainingWorkspaceSize(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        dataTensor,
        dataTensor,
        outTensor,
        scalebiasTensor,
        ActivationDesc,
        //NULL,
        scalebiasTensor,
        &workspace_size,
        1
    );
    // CHECK: reservespace_size = handle.get_batch_normalization_workspace_size(dpct::dnnl::batch_normalization_ops::none, dataTensor);
    cudnnGetNormalizationTrainingReserveSpaceSize(
        handle,
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        NULL,
        dataTensor,
        &reservespace_size,
        1
    );
    hipMalloc(&workspace, workspace_size);
    hipMalloc(&reservespace,  reservespace_size);
    // CHECK: handle.async_batch_normalization_forward_training(dpct::dnnl::batch_normalization_mode::spatial, dpct::dnnl::batch_normalization_ops::none, ActivationDesc, eps, factor, alpha, dataTensor, data, beta, outTensor, out, dataTensor, z, scalebiasTensor, scale, bias, scalebiasTensor, rmean, rvar, smean, svar, reservespace_size, reservespace);
    cudnnNormalizationForwardTraining(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        dataTensor,
        data,
        scalebiasTensor,
        scale,
        bias,
        factor,
        scalebiasTensor,
        rmean,
        rvar,
        eps,
        smean,
        svar,
        ActivationDesc,
        dataTensor,
        z,
        outTensor,
        out,
        workspace,
        workspace_size,
        reservespace,
        reservespace_size,
        1);


    float *bworkspace;
    size_t bworkspace_size;
    // CHECK: bworkspace_size = 0;
    cudnnGetNormalizationForwardTrainingWorkspaceSize(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        dataTensor,
        outTensor,
        outTensor,
        scalebiasTensor,
        ActivationDesc,
        scalebiasTensor,
        &bworkspace_size,
        1
    );
    hipMalloc(&bworkspace, bworkspace_size);
    // CHECK: handle.async_batch_normalization_backward(dpct::dnnl::batch_normalization_mode::spatial, dpct::dnnl::batch_normalization_ops::none, ActivationDesc, eps, alpha, dataTensor, data, outTensor, out, outTensor, diffout, beta, dataTensor, diffdata, outTensor, diffz, alpha, scalebiasTensor, scale, bias, beta, diffscale, diffbias, scalebiasTensor, smean, svar, reservespace_size, reservespace);
    cudnnNormalizationBackward(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        &alpha,
        &beta,
        dataTensor,
        data,
        outTensor,
        out,
        outTensor,
        diffout,
        outTensor,
        diffz,
        dataTensor,
        diffdata,
        scalebiasTensor,
        scale,
        bias,
        diffscale,
        diffbias,
        eps,
        scalebiasTensor,
        smean,
        svar,
        ActivationDesc,
        bworkspace,
        bworkspace_size,
        reservespace,
        reservespace_size,
        1);

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_smean.data(), smean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_svar.data(), svar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_diffdata.data(), diffdata, sizeof(float) * ele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_diffz.data(), diffz, sizeof(float) * ele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_diffscale.data(), diffscale,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_diffbias.data(), diffbias,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}