// RUN: dpct -in-root %S -out-root %T/lrn %S/lrn.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/lrn/lrn.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>
// CHECK: #include <dpct/dnnl_utils.hpp>
// CHECK: #include <iostream>
// CHECK: #include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

// CHECK: template <dpct::library_data_t T>
// CHECK: struct dt_trait {
// CHECK:     typedef void type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_float> {
// CHECK:     typedef float type;
// CHECK: };
// CHECK: template <>
// CHECK: /*
// CHECK: DPCT1007:{{[0-9]+}}: Migration of data type double is not supported.
// CHECK: */
// CHECK: struct dt_trait<CUDNN_DATA_DOUBLE> {
// CHECK:     typedef double type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_int32> {
// CHECK:     typedef int type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_half> {
// CHECK:     typedef float type;
// CHECK: };
template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};
template<>
struct dt_trait<CUDNN_DATA_DOUBLE>{
    typedef double type;
};
template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};

template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test1() {
    // CHECK: dpct::dnnl::engine_ext handle;
    // CHECK: dpct::dnnl::memory_desc_ext dataTensor, outTensor, diffdataTensor, diffoutTensor;

    // CHECK: handle.create_engine();

    // CHECK: sycl::queue *stream1;
    // CHECK: stream1 = dpct::get_current_device().create_queue();
    // CHECK: handle.set_queue(stream1);
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    //using HT = dt_trait<T>::type;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    HT *data, *out, *diffdata, *diffout;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num);
    std::vector<HT> host_diffdata(ele_num);
    std::vector<HT> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));
    hipMalloc(&diffdata, ele_num * sizeof(HT));
    hipMalloc(&diffout, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    unsigned int local_size = 3;
    float lrn_alpha = 1.5f;
    float lrn_beta = 1.5f;
    float lrn_k = 1.f;
    // CHECK: dpct::dnnl::lrn_desc desc;
    // CHECK: /*
    // CHECK: DPCT1026:{{[0-9]+}}: The call to cudnnCreateLRNDescriptor was removed because the function call is redundant in SYCL.
    // CHECK: */
    // CHECK: desc.set(local_size, lrn_alpha, lrn_beta, lrn_k);

    // CHECK: float alpha = 1.5f, beta = 0.f;
    // CHECK: handle.lrn_forward(desc, alpha, dataTensor, data, beta, outTensor, out);

    // CHECK: alpha = 2.f, beta = 0.f;
    // CHECK: dpct::get_current_device().queues_wait_and_throw();
    // CHECK: /*
    // CHECK: DPCT1097:{{[0-9]+}}: The function "lrn_backward" may require the workspace which is used to save intermediate results from the "lrn_forward". By default, a workspace from engine_ext is selected according to pointer of source data, but this may be error for workspace data race. You may need to rewrite this code.
    // CHECK: */
    // CHECK: /*
    // CHECK: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK: */
    // CHECK: auto s = (handle.lrn_backward(desc, alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, beta, diffdataTensor, diffdata), 0);
    cudnnLRNDescriptor_t desc;
    cudnnCreateLRNDescriptor(&desc);
    cudnnSetLRNDescriptor(desc, local_size, lrn_alpha, lrn_beta, lrn_k);

    float alpha = 1.5f, beta = 0.f;
    cudnnLRNCrossChannelForward(handle, desc, CUDNN_LRN_CROSS_CHANNEL_DIM1, &alpha, dataTensor, data, &beta, outTensor, out);

    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    auto s = cudnnLRNCrossChannelBackward(handle, desc, CUDNN_LRN_CROSS_CHANNEL_DIM1, &alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
}

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipSetDevice(1);
    
    test1<CUDNN_DATA_FLOAT>();

    return 0;
}