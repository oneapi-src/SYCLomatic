// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct -in-root %S -out-root %T/normtrain %S/normtrain.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/normtrain/normtrain.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 1, sbw = 1;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);


    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(oele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
        host_z[i] = 10;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * oele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);

    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.5f;
    // CHECK: dpct::dnnl::activation_desc ActivationDesc;
    // CHECK: ActivationDesc.set(dnnl::algorithm::eltwise_relu_use_dst_for_bwd, 0.0f);
    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_PROPAGATE_NAN, 0.0f);
 
    float *workspace, *reservespace;
    size_t workspace_size, reservespace_size;
    // CHECK: workspace_size = 0;
    cudnnGetNormalizationForwardTrainingWorkspaceSize(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        dataTensor,
        dataTensor,
        outTensor,
        scalebiasTensor,
        ActivationDesc,
        //NULL,
        scalebiasTensor,
        &workspace_size,
        1
    );
    // CHECK: reservespace_size = handle.get_batch_normalization_workspace_size(dpct::dnnl::batch_normalization_ops::none, dataTensor);
    cudnnGetNormalizationTrainingReserveSpaceSize(
        handle,
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        NULL,
        dataTensor,
        &reservespace_size,
        1
    );
    hipMalloc(&workspace, workspace_size);
    hipMalloc(&reservespace,  reservespace_size);
    // CHECK: auto status = DPCT_CHECK_ERROR(handle.async_batch_normalization_forward_training(dpct::dnnl::batch_normalization_mode::spatial, dpct::dnnl::batch_normalization_ops::none, ActivationDesc, eps, factor, alpha, dataTensor, data, beta, outTensor, out, dataTensor, z, scalebiasTensor, scale, bias, scalebiasTensor, rmean, rvar, smean, svar, reservespace_size, reservespace));
    auto status = cudnnNormalizationForwardTraining(
        handle, 
        //CUDNN_NORM_PER_ACTIVATION,
        CUDNN_NORM_PER_CHANNEL,
        CUDNN_NORM_OPS_NORM,
        //CUDNN_NORM_OPS_NORM_ACTIVATION,
        //CUDNN_NORM_OPS_NORM_ADD_ACTIVATION,
        CUDNN_NORM_ALGO_STANDARD,
        &alpha,
        &beta,
        dataTensor,
        data,
        scalebiasTensor,
        scale,
        bias,
        factor,
        scalebiasTensor,
        rmean,
        rvar,
        eps,
        smean,
        svar,
        ActivationDesc,
        dataTensor,
        z,
        outTensor,
        out,
        workspace,
        workspace_size,
        reservespace,
        reservespace_size,
        1);

    if(status == CUDNN_STATUS_SUCCESS) {
        std::cout << "success" << std::endl;
    } else {
        std::cout << "error" << std::endl;
        return 0;
    }

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_smean.data(), smean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_svar.data(), svar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}