// RUN: dpct -in-root %S -out-root %T/convolution %S/convolution.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/convolution/convolution.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/convolution/convolution.dp.cpp -o %T/convolution/convolution.dp.o %}
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor;
    cudnnFilterDescriptor_t filterTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateFilterDescriptor(&filterTensor);
    int in = 1, ic = 2, ih = 5, iw = 5;
    int on = 1, oc = 4, oh = 4, ow = 4;
    int fk = 4, fc = 2, fh = 2, fw = 2;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    
    int filterdim[4] = {fk, fc, fh, fw};
    // CHECK: filterTensor.set(dpct::dnnl::memory_format_tag::nchw, dpct::library_data_t::real_float, 4, filterdim);
    cudnnSetFilterNdDescriptor(filterTensor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 4, filterdim);
    
    float *data, *out, *filter, *bias;
    std::vector<float> host_data(in * ic * ih * iw, 1.0f);
    std::vector<float> host_out(on * oc * oh * ow, 0.0f);
    std::vector<float> host_bias(on * oc * oh * ow, 0.0f);
    std::vector<float> host_filter(fk * fc * fh * fw, 0.0f);


    hipMalloc(&data, sizeof(float) * in * ic * ih * iw);
    hipMalloc(&out, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&bias, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&filter, sizeof(float) * fk * fc * fh * fw);

    hipMemcpy(data, host_data.data(), sizeof(float) * in * ic * ih * iw, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(filter, host_filter.data(), sizeof(float) * fk * fc * fh * fw, hipMemcpyHostToDevice);
    // CHECK: dpct::dnnl::convolution_desc covdes;
    // CHECK: covdes.set(0, 0, 1, 1, 1, 1);
    // CHECK: covdes.set_group_count(2);
    cudnnConvolutionDescriptor_t covdes;
    cudnnCreateConvolutionDescriptor(&covdes);
    cudnnSetConvolution2dDescriptor(covdes, 0, 0, 1, 1, 1, 1, CUDNN_CONVOLUTION, CUDNN_DATA_FLOAT);
    cudnnSetConvolutionGroupCount(covdes, 2);

    // CHECK: covdes.get_forward_output_dim(dataTensor, filterTensor, &on, &oc, &oh, &ow);
    cudnnGetConvolution2dForwardOutputDim(covdes, dataTensor, filterTensor, &on, &oc, &oh, &ow);

    int out_dim[5];
    // CHECK: covdes.get_forward_output_dim(dataTensor, filterTensor, 5, out_dim);
    cudnnGetConvolutionNdForwardOutputDim(covdes, dataTensor, filterTensor, 5, out_dim);

    // CHECK: /*
    // CHECK: DPCT1007:{{[0-9]+}}: Migration of CUDNN_CONVOLUTION is not supported.
    // CHECK: */
    cudnnConvolutionMode_t mode = CUDNN_CONVOLUTION;
    int retCount;
    size_t size;
    void *workspacesize;
    // CHECK: size = 0;
    cudnnGetConvolutionForwardWorkspaceSize(handle, dataTensor, filterTensor, covdes, outTensor, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, &size);
    hipMalloc(&workspacesize, size);

    int dimo[4];
    // CHECK: covdes.get_forward_output_dim(dataTensor, filterTensor, 4, dimo);
    cudnnGetConvolutionNdForwardOutputDim(covdes, dataTensor, filterTensor, 4, dimo);

    float alpha = 1.0f, beta = 0.0f;
    // CHECK: handle.async_convolution_forward(covdes, dnnl::algorithm::convolution_direct, alpha, dataTensor, data, filterTensor, filter, beta, outTensor, out);
    cudnnConvolutionForward(handle, &alpha, dataTensor, data, filterTensor, filter, covdes, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, workspacesize, size, &beta, outTensor, out);
    // CHECK: handle.async_convolution_forward(covdes, dnnl::algorithm::convolution_direct, *(float *)(void *)&alpha, dataTensor, data, filterTensor, filter, *(float *)(void *)&beta, outTensor, out);
    cudnnConvolutionForward(handle, (void *)&alpha, dataTensor, data, filterTensor, filter, covdes, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, workspacesize, size, (void *)&beta, outTensor, out);
    // CHECK: handle.async_convolution_forward(covdes, dnnl::algorithm::convolution_direct, *(float *)&alpha, dataTensor, data, filterTensor, filter, *(float *)&beta, outTensor, out);
    cudnnConvolutionForward(handle, (float *)&alpha, dataTensor, data, filterTensor, filter, covdes, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, workspacesize, size, (float *)&beta, outTensor, out);

    hipDeviceSynchronize();
    hipMemcpy(host_bias.data(), bias, sizeof(float) * on * oc * oh * ow, hipMemcpyDeviceToHost);
    hipMemcpy(host_out.data(), out, sizeof(float) * on * oc * oh * ow, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}
