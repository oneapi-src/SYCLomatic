// RUN: dpct -in-root %S -out-root %T/bntrain %S/bntrain.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/bntrain/bntrain.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor, additionTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&scalebiasTensor);
    cudnnCreateTensorDescriptor(&additionTensor);

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 5, sbw = 5;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NHWC, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(scalebiasTensor, CUDNN_TENSOR_NHWC, CUDNN_DATA_FLOAT, sbn, sbc, sbh, sbw);

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;
    std::vector<float> host_data(ele_num, 1.0f);
    std::vector<float> host_z(ele_num, 1.0f);
    std::vector<float> host_out(oele_num, 0.0f);
    std::vector<float> host_scale(sele_num, 1.0f);
    std::vector<float> host_bias(sele_num, 0.0f);
    std::vector<float> host_rmean(sele_num, 0.0f);
    std::vector<float> host_rvar(sele_num, 0.0f);
    std::vector<float> host_smean(save * sele_num, 0.0f);
    std::vector<float> host_svar(save * sele_num, 0.0f);

    for(int i = 0; i < ele_num; i++) {
        host_data[i] =  i + 4.f;
        host_out[i] = 1.f;
    }
    for(int i = 0; i < sele_num; i++) {
        host_scale[i] = i;
        host_bias[i] = i;
        host_rmean[i] = i;
        host_rvar[i] = i;
        host_smean[i] = i;
        host_svar[i] = i;
    }

    hipMalloc(&data, sizeof(float) * ele_num);
    hipMalloc(&z, sizeof(float) * ele_num);
    hipMalloc(&out, sizeof(float) * oele_num);
    hipMalloc(&scale, sizeof(float) * sele_num);
    hipMalloc(&bias, sizeof(float) * sele_num);
    hipMalloc(&rmean, sizeof(float) * sele_num);
    hipMalloc(&rvar, sizeof(float) * sele_num);
    hipMalloc(&smean, sizeof(float) * save*sele_num);
    hipMalloc(&svar, sizeof(float)  * save*sele_num);

    hipMemcpy(data, host_data.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * ele_num, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * oele_num, hipMemcpyHostToDevice);
    hipMemcpy(scale, host_scale.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rmean, host_rmean.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(rvar, host_rvar.data(), sizeof(float) * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(smean, host_smean.data(),  sizeof(float) * save * sele_num, hipMemcpyHostToDevice);
    hipMemcpy(svar, host_svar.data(), sizeof(float) * save * sele_num, hipMemcpyHostToDevice);

    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.1f;
    // CHECK: auto status = (handle.async_batch_normalization_forward_training(dpct::dnnl::batch_normalization_mode::per_activation, eps, factor, alpha, dataTensor, data, beta, outTensor, out, scalebiasTensor, scale, bias, rmean, rvar, smean, svar), 0);
    auto status = cudnnBatchNormalizationForwardTraining(
        handle,
        CUDNN_BATCHNORM_PER_ACTIVATION,
        &alpha,
        &beta,
        dataTensor,
        data,
        outTensor,
        out,
        scalebiasTensor,
        scale,
        bias,
        factor,
        rmean,
        rvar,
        eps,
        smean,
        svar);


    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * oele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_smean.data(), smean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_svar.data(), svar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rmean.data(), rmean,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);
    hipMemcpy(host_rvar.data(), rvar,  sizeof(float) * save * sele_num, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}