// RUN: dpct -in-root %S -out-root %T/softmax %S/softmax.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/softmax/softmax.dp.cpp --match-full-lines %s

// CHECK: #include <CL/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>
// CHECK: #include <dpct/dnnl_utils.hpp>
// CHECK: #include <iostream>
// CHECK: #include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

// CHECK: template <dpct::library_data_t T>
// CHECK: struct dt_trait {
// CHECK:     typedef void type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_float> {
// CHECK:     typedef float type;
// CHECK: };
// CHECK: template <>
// CHECK: /*
// CHECK: DPCT1007:{{[0-9]+}}: Migration of data type double is not supported.
// CHECK: */
// CHECK: struct dt_trait<CUDNN_DATA_DOUBLE> {
// CHECK:     typedef double type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_int32> {
// CHECK:     typedef int type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_half> {
// CHECK:     typedef float type;
// CHECK: };

template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};
template<>
struct dt_trait<CUDNN_DATA_DOUBLE>{
    typedef double type;
};
template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};


template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test1() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    //using HT = dt_trait<T>::type;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    HT *data, *out, *diffdata, *diffout;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num);
    std::vector<HT> host_diffdata(ele_num);
    std::vector<HT> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num * sizeof(HT));
    hipMalloc(&diffdata, ele_num * sizeof(HT));
    hipMalloc(&diffout, ele_num * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);

    float alpha = 1.5f, beta = 0.f;
    // CHECK: handle.softmax_forward(dpct::dnnl::softmax_algorithm::normal, dpct::dnnl::softmax_mode::channel, alpha, dataTensor, data, beta, outTensor, out);
    // CHECK: dpct::get_default_queue().memcpy(host_out.data(), out, ele_num * sizeof(HT)).wait();
    // CHECK: alpha = 2.f, beta = 0.f;
    // CHECK: dpct::get_current_device().queues_wait_and_throw();
    // CHECK: /*
    // CHECK: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK: */
    // CHECK: auto s = (handle.softmax_backward(dpct::dnnl::softmax_algorithm::normal, dpct::dnnl::softmax_mode::channel, alpha, outTensor, out, diffoutTensor, diffout, beta, diffdataTensor, diffdata), 0);

    cudnnSoftmaxForward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, dataTensor, data, &beta, outTensor, out);
    hipMemcpy(host_out.data(), out, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    auto s = cudnnSoftmaxBackward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, outTensor, out, diffoutTensor, diffout, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
}

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipSetDevice(1);
    
    test1<CUDNN_DATA_FLOAT>();

    return 0;
}