// RUN: dpct -in-root %S -out-root %T/convolutionbackweight %S/convolutionbackweight.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/convolutionbackweight/convolutionbackweight.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor;
    cudnnTensorDescriptor_t diffdataTensor, diffoutTensor;
    cudnnFilterDescriptor_t filterTensor, difffilterTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateFilterDescriptor(&filterTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    cudnnCreateFilterDescriptor(&difffilterTensor);
    int in = 1, ic = 4, ih = 5, iw = 5;
    int on = 1, oc = 4, oh = 4, ow = 4;
    int fk = 4, fc = 2, fh = 2, fw = 2;
    int ele_num = in * ic * ih * iw;
    int oele_num = on * oc * oh * ow;
    int fele_num = fk *fc * fh * fw;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);

    int filterdim[4] = {fk, fc, fh, fw};
    cudnnSetFilterNdDescriptor(filterTensor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 4, filterdim);
    cudnnSetFilterNdDescriptor(difffilterTensor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NCHW, 4, filterdim);

    float *data, *out, *filter, *diffdata, *diffout, *difffilter;
    std::vector<float> host_data(in * ic * ih * iw, 1.0f);
    std::vector<float> host_out(on * oc * oh * ow, 0.0f);
    std::vector<float> host_filter(fk * fc * fh * fw, 0.0f);
    std::vector<float> host_diffdata(in * ic * ih * iw, 1.0f);
    std::vector<float> host_diffout(on * oc * oh * ow, 0.0f);
    std::vector<float> host_difffilter(fk * fc * fh * fw, 0.0f);

    hipMalloc(&data, sizeof(float) * in * ic * ih * iw);
    hipMalloc(&out, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&filter, sizeof(float) * fk * fc * fh * fw);
    hipMalloc(&diffdata, sizeof(float) * in * ic * ih * iw);
    hipMalloc(&diffout, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&difffilter, sizeof(float) * fk * fc * fh * fw);

    hipMemcpy(data, host_data.data(), sizeof(float) * in * ic * ih * iw, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(filter, host_filter.data(), sizeof(float) * fk * fc * fh * fw, hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), sizeof(float) * in * ic * ih * iw, hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(difffilter, host_difffilter.data(), sizeof(float) * fk * fc * fh * fw, hipMemcpyHostToDevice);

    cudnnConvolutionDescriptor_t covdes;
    cudnnCreateConvolutionDescriptor(&covdes);
    cudnnSetConvolution2dDescriptor(covdes, 0, 0, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);
    cudnnSetConvolutionGroupCount(covdes, 2);

    int retCount;

    size_t size;
    void *workspacesize;
    // CHECK: size = 0;
    cudnnGetConvolutionBackwardFilterWorkspaceSize(
        handle, 
        dataTensor,
        diffoutTensor, 
        covdes, 
        difffilterTensor, 
        CUDNN_CONVOLUTION_BWD_FILTER_ALGO_1, 
        &size);
    hipMalloc(&workspacesize, size);

    float alpha = 1.0f, beta = 0.f;
    // CHECK: handle.async_convolution_backward_weight(covdes, dnnl::algorithm::convolution_direct, alpha, dataTensor, data, diffoutTensor, diffout, beta, difffilterTensor, difffilter);
    cudnnConvolutionBackwardFilter(
        handle, 
        &alpha,
        dataTensor,
        data,
        diffoutTensor,
        diffout, 
        covdes, 
        CUDNN_CONVOLUTION_BWD_FILTER_ALGO_1,
        workspacesize, 
        size, 
        &beta, 
        difffilterTensor, 
        difffilter);
    hipDeviceSynchronize();
    hipMemcpy(host_difffilter.data(), difffilter, sizeof(float) * fele_num, hipMemcpyDeviceToHost);

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}