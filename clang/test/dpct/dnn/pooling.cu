// RUN: dpct -in-root %S -out-root %T/pooling %S/pooling.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/pooling/pooling.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/pooling/pooling.dp.cpp -o %T/pooling/pooling.dp.o %}

// CHECK: #include <dpct/dnnl_utils.hpp>
// CHECK: #include <sycl/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>
// CHECK: #include <iostream>
// CHECK: #include <vector>
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

// CHECK: template <dpct::library_data_t T>
// CHECK: struct dt_trait {
// CHECK:     typedef void type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_float> {
// CHECK:     typedef float type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_double> {
// CHECK:     typedef double type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_int32> {
// CHECK:     typedef int type;
// CHECK: };
// CHECK: template <>
// CHECK: struct dt_trait<dpct::library_data_t::real_half> {
// CHECK:     typedef float type;
// CHECK: };
template<cudnnDataType_t T>
struct dt_trait{
    typedef void type;
};
template<>
struct dt_trait<CUDNN_DATA_FLOAT>{
    typedef float type;
};
template<>
struct dt_trait<CUDNN_DATA_DOUBLE>{
    typedef double type;
};
template<>
struct dt_trait<CUDNN_DATA_INT32>{
    typedef int type;
};
template<>
struct dt_trait<CUDNN_DATA_HALF>{
    typedef float type;
};


template<cudnnDataType_t T, typename HT = typename dt_trait<T>::type>
void test1() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);
    // CHECK: dpct::dnnl::pooling_desc desc;
    // CHECK: /*
    // CHECK: DPCT1026:{{[0-9]+}}: The call to cudnnCreatePoolingDescriptor was removed because this call is redundant in SYCL.
    // CHECK: */
    // CHECK: /*
    // CHECK: DPCT1007:{{[0-9]+}}: Migration of Nan numbers propagation option is not supported.
    // CHECK: */
    // CHECK: desc.set(dnnl::algorithm::pooling_max, 4, 4, 3, 3, 2, 2);

    cudnnPoolingDescriptor_t desc;
    cudnnCreatePoolingDescriptor(&desc);
    cudnnSetPooling2dDescriptor(desc, CUDNN_POOLING_MAX, CUDNN_NOT_PROPAGATE_NAN, 4, 4, 3, 3, 2, 2);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;


    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);

    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, T, n, c, h, w);
    cudnnPoolingMode_t mode;
    cudnnNanPropagation_t pmode;
    int height, width, vpad, hpad, vstride, hstride;
    // CHECK: desc.get(&mode, &height, &width, &vpad, &hpad, &vstride, &hstride);
    cudnnGetPooling2dDescriptor(desc, &mode, &pmode, &height, &width, &vpad, &hpad, &vstride, &hstride);    
    
    int w_d[2], pad_d[2], stride_d[2], pndim;
    // CHECK: desc.get(2, &mode, &pndim, w_d, pad_d, stride_d);
    cudnnGetPoolingNdDescriptor(desc, 2, &mode, &pmode, &pndim, w_d, pad_d, stride_d);


    int on, oc, oh, ow;
    // CHECK: desc.get_forward_output_dim(dataTensor, &on, &oc, &oh, &ow);
    cudnnGetPooling2dForwardOutputDim(desc, dataTensor, &on, &oc, &oh, &ow);

    int out_dim[5];
    // CHECK: desc.get_forward_output_dim(dataTensor, 5, out_dim);
    cudnnGetPoolingNdForwardOutputDim(desc, dataTensor, 5, out_dim);

    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, T, on, oc, oh, ow);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, T, on, oc, oh, ow);
    int ele_num2 = on * oc * oh * ow;

    HT *data, *out, *diffdata, *diffout;
    std::vector<HT> host_data(ele_num);
    std::vector<HT> host_out(ele_num2);
    std::vector<HT> host_diffdata(ele_num);
    std::vector<HT> host_diffout(ele_num2);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_diffdata[i] = i;

    }
    for(int i = 0; i < ele_num2; i++) {
        host_out[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(HT));
    hipMalloc(&out, ele_num2 * sizeof(HT));
    hipMalloc(&diffdata, ele_num * sizeof(HT));
    hipMalloc(&diffout, ele_num2 * sizeof(HT));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num2 * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(HT), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num2 * sizeof(HT), hipMemcpyHostToDevice);

    float alpha = 1.5f, beta = 1.f;
    // CHECK: handle.async_pooling_forward(desc, alpha, dataTensor, data, beta, outTensor, out);
    // CHECK: dpct::get_in_order_queue().memcpy(host_out.data(), out, ele_num2 * sizeof(HT)).wait();
    // CHECK: dpct::get_current_device().queues_wait_and_throw();
    // CHECK: /*
    // CHECK: DPCT1097:{{[0-9]+}}: The function "async_pooling_backward" may require the workspace used to save intermediate results from function "async_pooling_forward". By default, a workspace from engine_ext is selected according to the source data pointer, but this may be incorrect and cause a workspace data race. You may need to rewrite this code.
    // CHECK: */
    // CHECK: auto s = DPCT_CHECK_ERROR(handle.async_pooling_backward(desc, alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, beta, diffdataTensor, diffdata));

    cudnnPoolingForward(handle, desc, &alpha, dataTensor, data, &beta, outTensor, out);
    hipMemcpy(host_out.data(), out, ele_num2 * sizeof(HT), hipMemcpyDeviceToHost);
    alpha = 1.5f, beta = 1.f;
    hipDeviceSynchronize();
    auto s = cudnnPoolingBackward(handle, desc, &alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(HT), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
}

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);
    hipSetDevice(1);
    
    test1<CUDNN_DATA_FLOAT>();

    return 0;
}