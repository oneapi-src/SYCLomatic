// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1, cuda-12.2
// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1, v12.2
// RUN: dpct -in-root %S -out-root %T/convolution_v7 %S/convolution_v7.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/convolution_v7/convolution_v7.dp.cpp --match-full-lines %s
#include <hip/hip_runtime.h>
#include <hipDNN.h>
#include <iostream>
#include <vector>

int main() {
    hipdnnHandle_t handle;
    hipdnnTensorDescriptor_t dataTensor;
    hipdnnFilterDescriptor_t filterTensor;
    hipdnnConvolutionDescriptor_t covdes;
    // CHECK: dnnl::algorithm bwd_filter_a;
    // CHECK: dnnl::algorithm bwd_data_a;
    // CHECK: dnnl::algorithm fwd_a;
    hipdnnConvolutionBwdFilterAlgo_t bwd_filter_a;
    hipdnnConvolutionBwdDataAlgo_t bwd_data_a;
    hipdnnConvolutionFwdAlgo_t fwd_a;
    // CHECK: bwd_filter_a = dnnl::algorithm::convolution_auto;
    hipdnnGetConvolutionBackwardFilterAlgorithm(handle, dataTensor, dataTensor, covdes, filterTensor, HIPDNN_CONVOLUTION_BWD_FILTER_PREFER_FASTEST, 10, &bwd_filter_a);
    // CHECK: bwd_data_a = dnnl::algorithm::convolution_auto;
    hipdnnGetConvolutionBackwardDataAlgorithm(handle, filterTensor, dataTensor, covdes, dataTensor, HIPDNN_CONVOLUTION_BWD_DATA_PREFER_FASTEST, 10, &bwd_data_a);    
    // CHECK: fwd_a = dnnl::algorithm::convolution_auto;
    hipdnnGetConvolutionForwardAlgorithm(handle, dataTensor, filterTensor, covdes, dataTensor, HIPDNN_CONVOLUTION_FWD_PREFER_FASTEST, 10, &fwd_a);

    return 0;
}
