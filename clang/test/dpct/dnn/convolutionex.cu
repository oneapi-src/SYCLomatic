// RUN: dpct -in-root %S -out-root %T/convolutionex %S/convolutionex.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/convolutionex/convolutionex.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/convolutionex/convolutionex.dp.cpp -o %T/convolutionex/convolutionex.dp.o %}

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    hipGetDeviceCount(&nDevices);

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, biasTensor;
    cudnnFilterDescriptor_t filterTensor;
    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateFilterDescriptor(&filterTensor);
    cudnnCreateTensorDescriptor(&biasTensor);
    int in = 1, ic = 2, ih = 5, iw = 5;
    int on = 1, oc = 4, oh = 4, ow = 4;
    int fk = 4, fc = 1, fh = 2, fw = 2;
    int ele_num = in * ic * ih * iw;
    int oele_num = on * oc * oh * ow;
    int fele_num = fk *fc * fh * fw;
    std::vector<int> bias_dim = {1, oc, 1, 1};
    std::vector<int> bias_stride = {oc, 1, 1, 1};
    int bele_num = oc * 1;
    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, in, ic, ih, iw);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, on, oc, oh, ow);
    cudnnSetTensorNdDescriptor(biasTensor, CUDNN_DATA_FLOAT, 4, bias_dim.data(), bias_stride.data());

    int filterdim[4] = {fk, fc, fh, fw};
    // CHECK: filterTensor.set(dpct::dnnl::memory_format_tag::nhwc, dpct::library_data_t::real_float, 4, filterdim);
    cudnnSetFilterNdDescriptor(filterTensor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NHWC, 4, filterdim);

    float *data, *out, *filter, *z, *bias;
    std::vector<float> host_data(in * ic * ih * iw, 1.0f);
    std::vector<float> host_out(on * oc * oh * ow, 0.0f);
    std::vector<float> host_z(oele_num, 0.0f);
    std::vector<float> host_bias(bele_num, 0.0f);
    std::vector<float> host_filter(fk * fc * fh * fw, 0.0f);

    for(int i = 0; i < in * ic * ih * iw; i++) {
        host_data[i] = i;
    }
    for(int i = 0; i < oele_num; i++) {
        host_out[i] = i;
        host_z[i] = i;
    }
    for(int i = 0; i < bele_num; i++) {
        host_bias[i] = i;
    }
    for(int i = 0; i < fele_num; i++) {
        host_filter[i] = i;
    }

    hipMalloc(&data, sizeof(float) * in * ic * ih * iw);
    hipMalloc(&out, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&z, sizeof(float) * on * oc * oh * ow);
    hipMalloc(&bias, sizeof(float) * bele_num);
    hipMalloc(&filter, sizeof(float) * fk * fc * fh * fw);

    hipMemcpy(data, host_data.data(), sizeof(float) * in * ic * ih * iw, hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(z, host_z.data(), sizeof(float) * on * oc * oh * ow, hipMemcpyHostToDevice);
    hipMemcpy(bias, host_bias.data(), sizeof(float) * bele_num, hipMemcpyHostToDevice);

    hipMemcpy(filter, host_filter.data(), sizeof(float) * fk * fc * fh * fw, hipMemcpyHostToDevice);

    cudnnConvolutionDescriptor_t covdes;
    cudnnCreateConvolutionDescriptor(&covdes);
    cudnnSetConvolution2dDescriptor(covdes, 0, 0, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);
    cudnnSetConvolutionGroupCount(covdes, 2);
    int retCount = 1;

    size_t size;
    void *workspacesize;
    cudnnGetConvolutionForwardWorkspaceSize(handle, dataTensor, filterTensor, covdes, outTensor, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, &size);
    hipMalloc(&workspacesize, size);

    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);

    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_NOT_PROPAGATE_NAN, 0.0f);
    

    float alpha = 1.5f, beta = 1.f;
    // CHECK: handle.async_convolution_forward(covdes, dnnl::algorithm::convolution_direct, ActivationDesc, alpha, dataTensor, data, filterTensor, filter, beta, outTensor, z, biasTensor, bias, outTensor, out);
    cudnnConvolutionBiasActivationForward(
        handle, 
        &alpha, 
        dataTensor, 
        data, 
        filterTensor, 
        filter, 
        covdes, 
        CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, 
        workspacesize, 
        size,
        &beta,
        outTensor,
        z,
        biasTensor,
        bias,
        ActivationDesc,
        outTensor, 
        out);

    hipDeviceSynchronize();
    hipMemcpy(host_out.data(), out, sizeof(float) * on * oc * oh * ow, hipMemcpyDeviceToHost);
    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    return 0;
}