// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --use-experimental-features=graph --format-range=none -out-root %T/cudaGraph_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cudaGraph_test/cudaGraph_test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cudaGraph_test/cudaGraph_test.dp.cpp -o %T/cudaGraph_test/cudaGraph_test.dp.o %}

#include <hip/hip_runtime.h>

int main() {
  // CHECK: dpct::experimental::command_graph_ptr graph;
  // CHECK-NEXT: dpct::experimental::command_graph_ptr *graph2;
  // CHECK-NEXT: dpct::experimental::command_graph_ptr **graph3;
  hipGraph_t graph;
  hipGraph_t *graph2;
  hipGraph_t **graph3;

  // CHECK: dpct::experimental::command_graph_ptr graph4[10];
  hipGraph_t graph4[10];

  // CHECK: dpct::experimental::command_graph_ptr graph5, *graph6, **graph7;
  hipGraph_t graph5, *graph6, **graph7;

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_ptr *execGraph2;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_ptr **execGraph3;
  hipGraphExec_t execGraph;
  hipGraphExec_t *execGraph2;
  hipGraphExec_t **execGraph3;

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph4[10];
  hipGraphExec_t execGraph4[10];

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph5, *execGraph6, **execGraph7;
  hipGraphExec_t execGraph5, *execGraph6, **execGraph7;

  // CHECK: execGraph = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>((*graph2)->finalize());
  // CHECK-NEXT: *execGraph2 = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>(graph->finalize());
  // CHECK-NEXT: **execGraph3 = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>((*graph2)->finalize());
  hipGraphInstantiate(&execGraph, *graph2, nullptr, nullptr, 0);
  hipGraphInstantiate(execGraph2, graph, nullptr, nullptr, 0);
  hipGraphInstantiate(*execGraph3, *graph2, nullptr, nullptr, 0);

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipStream_t *stream2;

  // CHECK: stream->ext_oneapi_graph(*execGraph);
  // CHECK-NEXT: (*stream2)->ext_oneapi_graph(**execGraph2);
  hipGraphLaunch(execGraph, stream);
  hipGraphLaunch(*execGraph2, *stream2);

  // CHECK: delete (execGraph);
  // CHECK-NEXT: delete (*execGraph2);
  // CHECK-NEXT:  delete (**execGraph3);
  hipGraphExecDestroy(execGraph);
  hipGraphExecDestroy(*execGraph2);
  hipGraphExecDestroy(**execGraph3);

  return 0;
}
