// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --use-experimental-features=graph --format-range=none -out-root %T/cudaGraph_test %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cudaGraph_test/cudaGraph_test.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -DBUILD_TEST -fsycl %T/cudaGraph_test/cudaGraph_test.dp.cpp -o %T/cudaGraph_test/cudaGraph_test.dp.o %}

#include <hip/hip_runtime.h>
#define CUDA_CHECK_THROW(x)  \
  do {                       \
    hipError_t _result = x; \
  } while (0)

int main() {
  // CHECK: dpct::experimental::command_graph_ptr graph;
  // CHECK-NEXT: dpct::experimental::command_graph_ptr *graph2;
  // CHECK-NEXT: dpct::experimental::command_graph_ptr **graph3;
  hipGraph_t graph;
  hipGraph_t *graph2;
  hipGraph_t **graph3;

  // CHECK: dpct::experimental::command_graph_ptr graph4[10];
  hipGraph_t graph4[10];

  // CHECK: dpct::experimental::command_graph_ptr graph5, *graph6, **graph7;
  hipGraph_t graph5, *graph6, **graph7;

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_ptr *execGraph2;
  // CHECK-NEXT: dpct::experimental::command_graph_exec_ptr **execGraph3;
  hipGraphExec_t execGraph;
  hipGraphExec_t *execGraph2;
  hipGraphExec_t **execGraph3;

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph4[10];
  hipGraphExec_t execGraph4[10];

  // CHECK: dpct::experimental::command_graph_exec_ptr execGraph5, *execGraph6, **execGraph7;
  hipGraphExec_t execGraph5, *execGraph6, **execGraph7;

  // CHECK: dpct::experimental::node_ptr node;
  // CHECK-NEXT: dpct::experimental::node_ptr *node2;
  // CHECK-NEXT: dpct::experimental::node_ptr **node3;
  hipGraphNode_t node;
  hipGraphNode_t *node2;
  hipGraphNode_t **node3;

  // CHECK: dpct::experimental::node_ptr node4[10];
  // CHECK-NEXT: dpct::experimental::node_ptr node5[10];
  hipGraphNode_t node4[10];
  hipGraphNode_t node5[10];

  // CHECK: dpct::experimental::node_ptr node6, *node7, **node8;
  hipGraphNode_t node6, *node7, **node8;

  // CHECK: const dpct::experimental::node_ptr node9 = nullptr;
  // CHECK-NEXT: const dpct::experimental::node_ptr node10[1] = {node};
  // CHECK-NEXT: const dpct::experimental::node_ptr node11[1] = {*node2};
  const hipGraphNode_t node9 = nullptr;
  const hipGraphNode_t node10[1] = {node};
  const hipGraphNode_t node11[1] = {*node2};

  // CHECK: dpct::experimental::add_empty_node(&node, graph, node4, 10);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(dpct::experimental::add_empty_node(&node, graph, node4, 10)));
  hipGraphAddEmptyNode(&node, graph, node4, 10);
  CUDA_CHECK_THROW(hipGraphAddEmptyNode(&node, graph, node4, 10));

  // CHECK: dpct::experimental::add_empty_node(node2, *graph2, NULL, 0);
  hipGraphAddEmptyNode(node2, *graph2, NULL, 0);

  // CHECK: dpct::experimental::add_empty_node(&node, graph, node10, 1);
  hipGraphAddEmptyNode(&node, graph, node10, 1);

  // CHECK: dpct::experimental::add_dependencies(graph, node4, node5, 10);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(dpct::experimental::add_dependencies(graph, node4, node5, 10)));
  hipGraphAddDependencies(graph, node4, node5, 10);
  CUDA_CHECK_THROW(hipGraphAddDependencies(graph, node4, node5, 10));

  // CHECK: dpct::experimental::add_dependencies(graph, node10, node11, 1);
  hipGraphAddDependencies(graph, node10, node11, 1);

  // CHECK: execGraph = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>((*graph2)->finalize());
  // CHECK-NEXT: *execGraph2 = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>(graph->finalize());
  // CHECK-NEXT: **execGraph3 = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>((*graph2)->finalize());
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(**execGraph3 = new sycl::ext::oneapi::experimental::command_graph<sycl::ext::oneapi::experimental::graph_state::executable>((*graph2)->finalize())));
  hipGraphInstantiate(&execGraph, *graph2, nullptr, nullptr, 0);
  hipGraphInstantiate(execGraph2, graph, nullptr, nullptr, 0);
  hipGraphInstantiate(*execGraph3, *graph2, nullptr, nullptr, 0);
  CUDA_CHECK_THROW(hipGraphInstantiate(*execGraph3, *graph2, nullptr, nullptr, 0));

  hipStream_t stream;
  hipStreamCreate(&stream);

  hipStream_t *stream2;

  // CHECK: stream->ext_oneapi_graph(*execGraph);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(stream->ext_oneapi_graph(*execGraph)));
  // CHECK-NEXT: (*stream2)->ext_oneapi_graph(**execGraph2);
  hipGraphLaunch(execGraph, stream);
  CUDA_CHECK_THROW(hipGraphLaunch(execGraph, stream));
  hipGraphLaunch(*execGraph2, *stream2);

#ifndef DBUILD_TEST
  // CHECK: execGraph->update(*graph);
  hipGraphExecUpdate(execGraph, graph, nullptr, nullptr);

  // CHECK: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(execGraph->update(*graph)));
  CUDA_CHECK_THROW(hipGraphExecUpdate(execGraph, graph, nullptr, nullptr));
#endif

  // CHECK: delete (execGraph);
  // CHECK-NEXT: delete (*execGraph2);
  // CHECK-NEXT:  delete (**execGraph3);
  // CHECK-NEXT: CUDA_CHECK_THROW(DPCT_CHECK_ERROR(delete (**execGraph3)));
  hipGraphExecDestroy(execGraph);
  hipGraphExecDestroy(*execGraph2);
  hipGraphExecDestroy(**execGraph3);
  CUDA_CHECK_THROW(hipGraphExecDestroy(**execGraph3));

  return 0;
}
