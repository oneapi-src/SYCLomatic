#include "hip/hip_runtime.h"
// RUN: dpct --no-dpcpp-extensions=enqueued_barriers --format-range=none -out-root %T/time-measure-usm-no-submit-barrier %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/time-measure-usm-no-submit-barrier/time-measure-usm-no-submit-barrier.dp.cpp --match-full-lines %s
#include <stdio.h>

#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    hipStream_t stream;

    int ha[N], hb[N];
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
    // CHECK: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


    hipEventCreate(&start);
    hipEventCreate(&stop);

   // CHECK:    sycl::event stop_q_ct1_1;
   // CHECK-NEXT:    sycl::event stop_q_ct1_2;
   // CHECK-NEXT:    sycl::event stop_stream_1;
   // CHECK-MEXT:    start_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);

    // CHECK: stop_q_ct1_1 = q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
    // CHECK: stop_q_ct1_2 = q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
    // CHECK: stop_stream_1 = stream->memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

    // CHECK: stop_stream_1.wait();
    // CHECK: stop_q_ct1_1.wait();
    // CHECK: stop_q_ct1_2.wait();
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    // CHECK: elapsedTime = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

    // CHECK: q_ct1.memcpy(hb, db, N*sizeof(int));
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}

#define SAFE_CALL(call)                                                   \
  do {                                                                         \
    int err = call;                                                            \
  } while (0)

void foo_usm() {
  hipStream_t s1, s2;
  int *gpu_t, *host_t, n = 10;
  hipEvent_t start, stop;
  SAFE_CALL(hipEventRecord(start, 0));

  // CHECK:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:SAFE_CALL((stop_s1_1 = s1->memcpy(gpu_t, host_t, n * sizeof(int)), 0));
  SAFE_CALL(hipMemcpyAsync(gpu_t, host_t, n * sizeof(int), hipMemcpyHostToDevice, s1));

  // CHECK:  DPCT1024:{{[0-9]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  stop_s1_1.wait();
  // CHECK-NEXT:  stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:  SAFE_CALL(0);
  // CHECK-NEXT:  SAFE_CALL(0);
  // CHECK-NEXT:  float Time = 0.0f;
  // CHECK-NEXT:  Time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
  SAFE_CALL(hipEventRecord(stop, 0));
  SAFE_CALL(hipEventSynchronize(stop));
  float Time = 0.0f;
  hipEventElapsedTime(&Time, start, stop);
}

__global__ void readTexels(int n, float *d_out, int width){}
__global__ void readTexelsFoo1(int n, float *d_out){}
__global__ void readTexelsFoo2(int n, float *d_out, int width, int height){}
texture<float4, 2, hipReadModeElementType> texA;

void foo()
{
    const unsigned int passes = 100;
    const unsigned int nsizes = 5;
    const unsigned int sizes[] = { 16, 64, 256, 1024, 4096 };
    const unsigned int kernelRepFoo[] = { 1024, 1024, 1024, 1024, 256 };
    const unsigned int iterations = 10;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int j = 0; j < nsizes; j++)
    {
        const unsigned int size      = 1024 * sizes[j];
        const unsigned int numFloat  = size / sizeof(float);
        const unsigned int numFloat4 = size / sizeof(float4);
        size_t width, height;
        const unsigned int kernelRepFactor = kernelRepFoo[j];

        // Image memory sizes should be power of 2.
        size_t sizeLog = lround(log2(double(numFloat4)));
        height = 1 << (sizeLog >> 1);  // height is the smaller size
        width = numFloat4 / height;

        const dim3 blockSize(16, 8);
        const dim3 gridSize(width/blockSize.x, height/blockSize.y);

        float *h_in = new float[numFloat];
        float *h_out = new float[numFloat4];
        float *d_out;
        hipMalloc((void**) &d_out, numFloat4 * sizeof(float));

        // Allocate a cuda array
        hipArray* cuArray;
        hipMallocArray(&cuArray, &texA.channelDesc, width, height);

        // Copy in source data
        hipMemcpyToArray(cuArray, 0, 0, h_in, size, hipMemcpyHostToDevice);

        // Bind texture to the array
        hipBindTextureToArray(texA, cuArray);

        for (int p = 0; p < passes; p++)
        {
            // Test 1: Repeated Linear Access
            float t = 0.0f;

            hipEventRecord(start, 0);
            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {
// CHECK:                 DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                 */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexels_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexels(kernelRepFactor, d_out, width);
// CHECK-NEXT:                      });
                readTexels<<<gridSize, blockSize>>>(kernelRepFactor, d_out,
                                                    width);
            }

// CHECK:             DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:            */
// CHECK-NEXT:            dpct::get_current_device().queues_wait_and_throw();
// CHECK-NEXT:            stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:            t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 2 Repeated Cache Access
            hipEventRecord(start, 0);
            for (int iter = 0; iter < iterations; iter++)
            {

// CHECK:                DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexelsFoo1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexelsFoo1(kernelRepFactor, d_out);
// CHECK-NEXT:                      });
                readTexelsFoo1<<<gridSize, blockSize>>>
                        (kernelRepFactor, d_out);
            }

// CHECK:             DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:            */
// CHECK-NEXT:            dpct::get_current_device().queues_wait_and_throw();
// CHECK-NEXT:            stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:            t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 3 Repeated "Random" Access
            hipEventRecord(start, 0);

            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {

// CHECK:                DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexelsFoo2_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexelsFoo2(kernelRepFactor, d_out, width, height);
// CHECK-NEXT:                      });
                readTexelsFoo2<<<gridSize, blockSize>>>
                                (kernelRepFactor, d_out, width, height);
            }

// CHECK:             DPCT1012:{{[0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
// CHECK-NEXT:            */
// CHECK-NEXT:            dpct::get_current_device().queues_wait_and_throw();
// CHECK-NEXT:            stop_ct1 = std::chrono::steady_clock::now();
// CHECK-NEXT:            t = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

        }
        delete[] h_in;
        delete[] h_out;
        hipFree(d_out);
        hipFreeArray(cuArray);
        hipUnbindTexture(texA);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


__global__ void kernelFunc(){}

void barr(int maxCalls) {
  hipEvent_t evtStart[maxCalls];
  hipEvent_t evtEnd[maxCalls];
  float time[maxCalls];
  for (int i = 0; i < maxCalls; i++) {
    hipEventCreate( &(evtStart[i]) );
    hipEventCreate( &(evtEnd[i]) );
    time[i] = 0.0;
  }

  // CHECK: evtStart_ct1[0] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[0], 0 );
  // CHECK: evtEnd[0]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[0] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[0], 0 );

  // CHECK: evtStart_ct1[1] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[1], 0 );
  // CHECK: evtEnd[1]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[1] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[1], 0 );

  // CHECK: evtStart_ct1[2] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[2], 0 );
  // CHECK: evtEnd[2]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[2] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[2], 0 );

  // CHECK: dev_ct1.queues_wait_and_throw();
  hipDeviceSynchronize();

  float total;
  int i=0;
  hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
  float timesum = 0.0f;
  for (int i = 1; i < maxCalls; i++) {
    hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
    timesum += time[i];
  }
  hipEventElapsedTime( &total, evtStart[1], evtEnd[maxCalls-1]);
}

template <class T, int blockSize>
__global__ void
reduce(const T* __restrict__ g_idata, T* __restrict__ g_odata,
       int n) {}

template <class T, class vecT>
void RunTest()
{
    int probSizes[4] = { 1, 8, 32, 64 };
    int size;
    // Convert to MiB
    size = (size * 1024 * 1024) / sizeof(T);
    // create input data on CPU
    unsigned int bytes = size * sizeof(T);

    // Allocate Host Memory
    T* h_idata;
    T* reference;
    T* h_odata;

    int num_blocks  = 64;
    int num_threads = 256;
    int smem_size = sizeof(T) * num_threads;

    // Allocate device memory
    T* d_idata, *d_odata, *d_block_sums;
    hipEvent_t start, stop;
    int passes;
    int iters;

    for (int k = 0; k < passes; k++)
    {
        float totalScanTime = 0.0f;
        SAFE_CALL(hipEventRecord(start, 0));
        for (int j = 0; j < iters; j++)
        {
// CHECK:            q_ct1.parallel_for<dpct_kernel_name<class reduce_{{[a-z0-9]+}}, T, dpct_kernel_scalar<256>>>(
// CHECK-NEXT:                  sycl::nd_range<3>(sycl::range<3>(1, 1, num_blocks) * sycl::range<3>(1, 1, num_threads), sycl::range<3>(1, 1, num_threads)),
// CHECK-NEXT:                  [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                    reduce<T, 256>(d_idata, d_block_sums, size);
// CHECK-NEXT:                  });
            reduce<T, 256><<<num_blocks, num_threads, smem_size>>>(d_idata, d_block_sums, size);
        }
        SAFE_CALL(hipEventRecord(stop, 0));
        SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&totalScanTime, start, stop);
    }
    SAFE_CALL(hipFree(d_idata));
    SAFE_CALL(hipFree(d_odata));
    SAFE_CALL(hipFree(d_block_sums));
    SAFE_CALL(hipHostFree(h_idata));
    SAFE_CALL(hipHostFree(h_odata));
    SAFE_CALL(hipHostFree(reference));
    SAFE_CALL(hipEventDestroy(start));
    SAFE_CALL(hipEventDestroy(stop));
}

int foo_test_5() {
   RunTest<float, float4>();
}

__global__ void foo_kernel_1(unsigned short* blk_sad, unsigned short* frame,
                            int mb_width, int mb_height,
                            unsigned short* img_ref) {}

__global__ void foo_kernel_2(unsigned short* blk_sad, int mb_width,
                                  int mb_height) {}

__global__ void foo_kernel_3(unsigned short* blk_sad, int mb_width,
                                   int mb_height) {}

void test_1999(void* ref_image, void* cur_image,
                    float* sad_calc_ms, float* sad_calc_8_ms,
                    float* sad_calc_16_ms,
                    unsigned short** h_sads) {
    size_t image_width_macroblocks;
    size_t image_height_macroblocks;
    size_t image_size_macroblocks;
    size_t nsads;
    unsigned short* imgRef = NULL;
    unsigned short* d_cur_image = NULL;
    unsigned short* d_sads = NULL;

// CHECK:     dpct::event_ptr sad_calc_start, sad_calc_stop;
// CHECK-NEXT:     std::chrono::time_point<std::chrono::steady_clock> sad_calc_start_ct1;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> sad_calc_stop_ct1;
    hipEvent_t sad_calc_start, sad_calc_stop;
    hipEventCreate(&sad_calc_start);
    hipEventCreate(&sad_calc_stop);
    hipEventRecord(sad_calc_start);
    dim3 foo_kernel_1_threads_in_block;
    dim3 foo_kernel_1_blocks_in_grid;

// CHECK:    *sad_calc_stop = q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_1_blocks_in_grid * foo_kernel_1_threads_in_block, foo_kernel_1_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_1(d_sads, d_cur_image, image_width_macroblocks, image_height_macroblocks, imgRef);
// CHECK-NEXT:          });
    foo_kernel_1<<<foo_kernel_1_blocks_in_grid,
                  foo_kernel_1_threads_in_block>>>(d_sads, d_cur_image,
                                                  image_width_macroblocks,
                                                  image_height_macroblocks,
                                                  imgRef);

// CHECK:    sad_calc_stop->wait();
// CHECK-NEXT:    sad_calc_stop_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(sad_calc_stop);

// CHECK:    dpct::event_ptr sad_calc_8_start, sad_calc_8_stop;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> sad_calc_8_start_ct1;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> sad_calc_8_stop_ct1;
    hipEvent_t sad_calc_8_start, sad_calc_8_stop;

    hipEventCreate(&sad_calc_8_start);
    hipEventCreate(&sad_calc_8_stop);
    hipEventRecord(sad_calc_8_start);
    dim3 foo_kernel_2_threads_in_block;
    dim3 foo_kernel_2_blocks_in_grid;

// CHECK:    *sad_calc_8_stop = q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_2_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_2_blocks_in_grid * foo_kernel_2_threads_in_block, foo_kernel_2_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_2(d_sads, image_width_macroblocks, image_height_macroblocks);
// CHECK-NEXT:          });
    foo_kernel_2<<<
      foo_kernel_2_blocks_in_grid,
      foo_kernel_2_threads_in_block>>>(d_sads, image_width_macroblocks,
                                            image_height_macroblocks);
// CHECK:    sad_calc_8_stop->wait();
// CHECK-NEXT:    sad_calc_8_stop_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(sad_calc_8_stop);


// CHECK:    dpct::event_ptr sad_calc_16_start, sad_calc_16_stop;
// CHECK-NEXT:    std::chrono::time_point<std::chrono::steady_clock> sad_calc_16_start_ct1;
    hipEvent_t sad_calc_16_start, sad_calc_16_stop;

    hipEventCreate(&sad_calc_16_start);
    hipEventCreate(&sad_calc_16_stop);
    hipEventRecord(sad_calc_16_start);
    dim3 foo_kernel_3_threads_in_block;
    dim3 foo_kernel_3_blocks_in_grid;

// CHECK:    *sad_calc_16_stop = q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_3_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_3_blocks_in_grid * foo_kernel_3_threads_in_block, foo_kernel_3_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_3(d_sads, image_width_macroblocks, image_height_macroblocks);
// CHECK-NEXT:          });
    foo_kernel_3<<<
      foo_kernel_3_blocks_in_grid,
      foo_kernel_3_threads_in_block>>>(d_sads, image_width_macroblocks,
                                             image_height_macroblocks);
// CHECK:    sad_calc_16_stop->wait();
// CHECK-NEXT:    sad_calc_16_stop_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(sad_calc_16_stop);

    hipHostMalloc((void **)h_sads, nsads * sizeof(unsigned short));
    hipMemcpy(*h_sads, d_sads, nsads * sizeof(*d_sads), hipMemcpyDeviceToHost);
    hipFree(d_sads);
    hipFree(d_cur_image);
    hipFree(imgRef);

// CHECK:    *(sad_calc_ms) = std::chrono::duration<float, std::milli>(sad_calc_stop_ct1 - sad_calc_start_ct1).count();
// CHECK-NEXT:    *(sad_calc_8_ms) = std::chrono::duration<float, std::milli>(sad_calc_8_stop_ct1 - sad_calc_8_start_ct1).count();
// CHECK-NEXT:    *(sad_calc_16_ms) = std::chrono::duration<float, std::milli>(sad_calc_16_stop_ct1 - sad_calc_16_start_ct1).count();
    hipEventElapsedTime(sad_calc_ms, sad_calc_start, sad_calc_stop);
    hipEventElapsedTime(sad_calc_8_ms, sad_calc_8_start, sad_calc_8_stop);
    hipEventElapsedTime(sad_calc_16_ms, sad_calc_16_start, sad_calc_16_stop);
}

class Stream {};
namespace c10 {
namespace cuda {
class CUDAStream {
public:
  operator hipStream_t() const {
    hipStream_t tt;
    return tt;
  }
};
CUDAStream getCurrentCUDAStream(int device_index = -1);
} // namespace cuda
} // namespace c10


void foo() {
  c10::cuda::CUDAStream stream = c10::cuda::getCurrentCUDAStream();
  void* dst = NULL;
  void* src = NULL;
  int nbytes;
  hipMemcpyKind kind;

  // CHECK:  ((sycl::queue*)(stream))->memcpy(dst, src, nbytes);
  // CHECK-NEXT:  ((sycl::queue*)(stream))->memset(dst, 0, nbytes);
  hipMemcpyAsync(dst, src, nbytes, kind, stream);
  hipMemsetAsync(dst, 0, nbytes, stream);
}

