//RUN: dpct -out-root %T/curand-cross-function-placeholder %s --format-range=none --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
//RUN: FileCheck --input-file %T/curand-cross-function-placeholder/curand-cross-function-placeholder.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

//CHECK: void update(float* randvals, dpct::rng::host_rng_ptr rng, long long nx, long long ny) {
//CHECK-NEXT:   rng->generate_uniform(randvals, nx*ny/2);
//CHECK-NEXT: }
void update(float* randvals, hiprandGenerator_t rng, long long nx, long long ny) {
  hiprandGenerateUniform(rng, randvals, nx*ny/2);
}


//CHECK: int main(){
//CHECK-NEXT:   long long nx = 5120;
//CHECK-NEXT:   long long ny = 5120;
//CHECK-NEXT:   unsigned long long seed = 1234ULL;
//CHECK-NEXT:   dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:   dpct::rng::host_rng_ptr rng1;
//CHECK-NEXT:   rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::philox4x32x10);
//CHECK-NEXT:   rng1 = dpct::rng::create_host_rng(dpct::rng::random_engine_type::mrg32k3a);
//CHECK-NEXT:   rng->set_seed(seed);
//CHECK-NEXT:   rng1->set_seed(seed);
//CHECK-NEXT:   float *randvals;
//CHECK-NEXT:   rng->generate_uniform(randvals, nx*ny/2);
//CHECK-NEXT:   update(randvals, rng1, nx, ny);
//CHECK-NEXT:   rng.reset();
//CHECK-NEXT:   rng1.reset();
//CHECK-NEXT: }
int main(){
  long long nx = 5120;
  long long ny = 5120;
  unsigned long long seed = 1234ULL;
  hiprandGenerator_t rng;
  hiprandGenerator_t rng1;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandCreateGenerator(&rng1, HIPRAND_RNG_PSEUDO_MRG32K3A);
  hiprandSetPseudoRandomGeneratorSeed(rng, seed);
  hiprandSetPseudoRandomGeneratorSeed(rng1, seed);
  float *randvals;
  hiprandGenerateUniform(rng, randvals, nx*ny/2);
  update(randvals, rng1, nx, ny);
  hiprandDestroyGenerator(rng);
  hiprandDestroyGenerator(rng1);
}

//CHECK: void foo(){
//CHECK-NEXT:   float *randvals;
//CHECK-NEXT:   dpct::rng::host_rng_ptr rng;
//CHECK-NEXT:   rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::philox4x32x10);
//CHECK-NEXT:   rng->set_seed(111);
//CHECK-NEXT:   rng->generate_uniform(randvals, 0);
//CHECK-EMPTY:
//CHECK-NEXT:   rng = dpct::rng::create_host_rng(dpct::rng::random_engine_type::mrg32k3a);
//CHECK-NEXT:   rng->set_seed(222);
//CHECK-NEXT:   rng->generate_uniform(randvals, 0);
//CHECK-NEXT:   rng.reset();
//CHECK-NEXT: }
void foo(){
  float *randvals;
  hiprandGenerator_t rng;
  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  hiprandSetPseudoRandomGeneratorSeed(rng, 111);
  hiprandGenerateUniform(rng, randvals, 0);

  hiprandCreateGenerator(&rng, HIPRAND_RNG_PSEUDO_MRG32K3A);
  hiprandSetPseudoRandomGeneratorSeed(rng, 222);
  hiprandGenerateUniform(rng, randvals, 0);
  hiprandDestroyGenerator(rng);
}

