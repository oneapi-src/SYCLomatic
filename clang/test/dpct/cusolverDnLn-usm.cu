// RUN: dpct --format-range=none -out-root %T/cusolverDnLn-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn-usm/cusolverDnLn-usm.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusolverDnLn-usm/cusolverDnLn-usm.dp.cpp -o %T/cusolverDnLn-usm/cusolverDnLn-usm.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    float C_f = 0;
    double C_d = 0.0;
    hipComplex C_c = make_hipComplex(1,0);
    hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    size_t b_size;

    //CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, (float*)&A_f, lda, (float*)&workspace_f, Lwork));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, (double*)&A_d, lda, (double*)&workspace_d, Lwork));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&workspace_c, Lwork));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potrf(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&workspace_z, Lwork));
    status = hipsolverDnSpotrf(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    status = hipsolverDnDpotrf(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    status = hipsolverDnCpotrf(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    status = hipsolverDnZpotrf(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<float>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: float *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<float>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, (float*)&C_f, lda, (float*)&B_f, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<double>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: double *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<double>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, (double*)&C_d, lda, (double*)&B_d, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<float>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: std::complex<float> *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<std::complex<float>>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, (std::complex<float>*)&C_c, lda, (std::complex<float>*)&B_c, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs_scratchpad_size<std::complex<double>>(**cusolverH ,uplo ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: std::complex<double> *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<std::complex<double>>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::potrs(**cusolverH, uplo, n, nrhs, (std::complex<double>*)&C_z, lda, (std::complex<double>*)&B_z, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    hipsolverDnSpotrs(*cusolverH, uplo, n, nrhs, &C_f, lda, &B_f, ldb, &devInfo);
    hipsolverDnDpotrs(*cusolverH, uplo, n, nrhs, &C_d, lda, &B_d, ldb, &devInfo);
    hipsolverDnCpotrs(*cusolverH, uplo, n, nrhs, &C_c, lda, &B_c, ldb, &devInfo);
    hipsolverDnZpotrs(*cusolverH, uplo, n, nrhs, &C_z, lda, &B_z, ldb, &devInfo);

    //CHECK:  {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<float>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnSgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, (float*)&A_f, lda, &result_temp_pointer6, (float*)&workspace_f, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<double>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnDgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, (double*)&A_d, lda, &result_temp_pointer6, (double*)&workspace_d, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<float>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnCgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, (std::complex<float>*)&A_c, lda, &result_temp_pointer6, (std::complex<float>*)&workspace_c, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrf_scratchpad_size<std::complex<double>>(**cusolverH ,m ,n ,lda);
    //CHECK-NEXT: /*
    //CHECK-NEXT: DPCT1047:{{[0-9]+}}: The meaning of &devIpiv in the oneapi::mkl::lapack::getrf is different from the cusolverDnZgetrf. You may need to check the migrated code.
    //CHECK-NEXT: */
    //CHECK-NEXT: oneapi::mkl::lapack::getrf(**cusolverH, m, n, (std::complex<double>*)&A_z, lda, &result_temp_pointer6, (std::complex<double>*)&workspace_z, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: }
    hipsolverDnSgetrf(*cusolverH, m, n, &A_f, lda, &workspace_f, &devIpiv, &devInfo);
    hipsolverDnDgetrf(*cusolverH, m, n, &A_d, lda, &workspace_d, &devIpiv, &devInfo);
    hipsolverDnCgetrf(*cusolverH, m, n, &A_c, lda, &workspace_c, &devIpiv, &devInfo);
    hipsolverDnZgetrf(*cusolverH, m, n, &A_z, lda, &workspace_z, &devIpiv, &devInfo);


    //CHECK: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<float>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: float *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<float>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, (float*)&A_f, lda, &result_temp_pointer6, (float*)&B_f, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<double>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: double *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<double>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, (double*)&A_d, lda, &result_temp_pointer6, (double*)&B_d, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<float>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: std::complex<float> *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<std::complex<float>>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, (std::complex<float>*)&A_c, lda, &result_temp_pointer6, (std::complex<float>*)&B_c, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer6;
    //CHECK-NEXT: std::int64_t scratchpad_size_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs_scratchpad_size<std::complex<double>>(**cusolverH ,trans ,n ,nrhs ,lda ,ldb);
    //CHECK-NEXT: std::complex<double> *scratchpad_ct{{[0-9]+}} = sycl::malloc_device<std::complex<double>>(scratchpad_size_ct{{[0-9]+}}, **cusolverH);
    //CHECK-NEXT: sycl::event event_ct{{[0-9]+}};
    //CHECK-NEXT: event_ct{{[0-9]+}} = oneapi::mkl::lapack::getrs(**cusolverH, trans, n, nrhs, (std::complex<double>*)&A_z, lda, &result_temp_pointer6, (std::complex<double>*)&B_z, ldb, scratchpad_ct{{[0-9]+}}, scratchpad_size_ct{{[0-9]+}});
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer6;
    //CHECK-NEXT: std::vector<void *> ws_vec_ct{{[0-9]+}}{scratchpad_ct{{[0-9]+}}};
    //CHECK-NEXT: dpct::async_dpct_free(ws_vec_ct{{[0-9]+}}, {event_ct{{[0-9]+}}}, **cusolverH);
    //CHECK-NEXT: }
    hipsolverDnSgetrs(*cusolverH, trans, n, nrhs, &A_f, lda, &devIpiv, &B_f, ldb, &devInfo);
    hipsolverDnDgetrs(*cusolverH, trans, n, nrhs, &A_d, lda, &devIpiv, &B_d, ldb, &devInfo);
    hipsolverDnCgetrs(*cusolverH, trans, n, nrhs, &A_c, lda, &devIpiv, &B_c, ldb, &devInfo);
    hipsolverDnZgetrs(*cusolverH, trans, n, nrhs, &A_z, lda, &devIpiv, &B_z, ldb, &devInfo);


    //CHECK: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, (float*)&A_f, lda, (float*)&TAU_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, (double*)&A_d, lda, (double*)&TAU_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::geqrf(**cusolverH, m, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSgeqrf(*cusolverH, m, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDgeqrf(*cusolverH, m, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnCgeqrf(*cusolverH, m, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZgeqrf(*cusolverH, m, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    //CHECK: oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, (float*)&A_f, lda, (float*)&TAU_f, (float*)&B_f, ldb, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ormqr(**cusolverH, side, trans, m, n, k, (double*)&A_d, lda, (double*)&TAU_d, (double*)&B_d, ldb, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&B_c, ldb, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::unmqr(**cusolverH, side, trans, m, n, k, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&B_z, ldb, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSormqr(*cusolverH, side, trans, m, n, k, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnDormqr(*cusolverH, side, trans, m, n, k, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnCunmqr(*cusolverH, side, trans, m, n, k, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnZunmqr(*cusolverH, side, trans, m, n, k, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);


    //CHECK: oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, (float*)&A_f, lda, (float*)&TAU_f, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::orgqr(**cusolverH, m, n, k, (double*)&A_d, lda, (double*)&TAU_d, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&TAU_c, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::ungqr(**cusolverH, m, n, k, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&TAU_z, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSorgqr(*cusolverH, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnDorgqr(*cusolverH, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnCungqr(*cusolverH, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnZungqr(*cusolverH, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: int64_t result_temp_pointer5;
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, (float*)&A_f, lda, &result_temp_pointer5, (float*)&workspace_f, Lwork);
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer5;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer5;
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, (double*)&A_d, lda, &result_temp_pointer5, (double*)&workspace_d, Lwork);
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer5;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer5;
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, &result_temp_pointer5, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer5;
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: int64_t result_temp_pointer5;
    //CHECK-NEXT: oneapi::mkl::lapack::sytrf(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, &result_temp_pointer5, (std::complex<double>*)&workspace_z, Lwork);
    //CHECK-NEXT:  *&devIpiv = result_temp_pointer5;
    //CHECK-NEXT: }
    hipsolverDnSsytrf(*cusolverH, uplo, n, &A_f, lda, &devIpiv, &workspace_f, Lwork, &devInfo);
    hipsolverDnDsytrf(*cusolverH, uplo, n, &A_d, lda, &devIpiv, &workspace_d, Lwork, &devInfo);
    hipsolverDnCsytrf(*cusolverH, uplo, n, &A_c, lda, &devIpiv, &workspace_c, Lwork, &devInfo);
    hipsolverDnZsytrf(*cusolverH, uplo, n, &A_z, lda, &devIpiv, &workspace_z, Lwork, &devInfo);
}

