#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/cuda-math-intrinsics %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cuda-math-intrinsics/cuda-math-intrinsics.dp.cpp --match-full-lines %s

#include <cmath>
#include <iomanip>
#include <iostream>
#include <limits>
#include <algorithm>
#include <complex>

#include <stdio.h>

// CHECK: #include <algorithm>

#include "hip/hip_fp16.h"

using namespace std;

// CHECK: using sycl::max;
using ::max;

// CHECK: dpct::constant_memory<double, 0> d;
// CHECK-NEXT: dpct::constant_memory<double, 0> d2;
__constant__ double d;
__constant__ double d2;

// CHECK: double test(double d3, double d) {
// CHECK-NEXT:  return sycl::max(d, d3);
// CHECK-NEXT:}
__device__ double test(double d3) {
  return max(d, d3);
}

// CHECK:  double test2(double d, double d2) {
// CHECK-NEXT:   return sycl::max(d, d2);
// CHECK-NEXT: }
__device__ double test2() {
  return max(d, d2);
}

// CHECK:  double test3(double d4, double d5) {
// CHECK-NEXT:   return sycl::max(d4, d5);
// CHECK-NEXT: }
__device__ double test3(double d4, double d5) {
  return max(d4, d5);
}

// CHECK: dpct::constant_memory<float, 0> C;
// CHECK-NEXT:  int foo(int n, float C) {
// CHECK-NEXT:   return n == 1 ? C : 0;
// CHECK-NEXT: }
__constant__ float C;
__device__ int foo(int n) {
  return n == 1 ? C : 0;
}

__global__ void kernelFuncHalf(double *deviceArrayDouble) {
  __half h, h_1, h_2;
  __half2 h2, h2_1, h2_2;
  bool b;

  // Half Arithmetic Functions

  // TODO:1CHECK: h2_2 = h2 / h2_1;
  //h2_2 = __h2div(h2, h2_1);
  // TODO:1CHECK: h_2 = h / h_1;
  //h_2 = __hdiv(h, h_1);
  // CHECK: h_2 = sycl::fma(h, h_1, h_2);
  h_2 = __hfma(h, h_1, h_2);
  // CHECK: h_2 = h * h_1;
  h_2 = __hmul(h, h_1);
  // CHECK: h_2 = -h;
  h_2 = __hneg(h);
  // CHECK: h_2 = h - h_1;
  h_2 = __hsub(h, h_1);

  // Half2 Arithmetic Functions

  // CHECK: h2_2 = h2 + h2_1;
  h2_2 = __hadd2(h2, h2_1);
  // CHECK: h2_2 = sycl::fma(h2, h2_1, h2_2);
  h2_2 = __hfma2(h2, h2_1, h2_2);
  // CHECK: h2_2 = h2 * h2_1;
  h2_2 = __hmul2(h2, h2_1);
  // CHECK: h2_2 = -h2;
  h2_2 = __hneg2(h2);
  // CHECK: h2_2 = h2 - h2_1;
  h2_2 = __hsub2(h2, h2_1);

  // Half Comparison Functions

  // CHECK: b = h == h_1;
  b = __heq(h, h_1);
  // CHECK: b = h >= h_1;
  b = __hge(h, h_1);
  // CHECK: b = h > h_1;
  b = __hgt(h, h_1);
  // CHECK: b = sycl::isinf(h);
  b = __hisinf(h);
  // CHECK: b = sycl::isnan(h);
  b = __hisnan(h);
  // CHECK: b = h <= h_1;
  b = __hle(h, h_1);
  // CHECK: b = h < h_1;
  b = __hlt(h, h_1);
  // CHECK: b = h != h_1;
  b = __hne(h, h_1);

  // Half2 Comparison Functions

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __heq2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __heq2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hge2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hge2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hgt2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hgt2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hisnan2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hisnan2(h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hle2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hle2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hlt2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hlt2(h2, h2_1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hne2 is not supported.
  // CHECK-NEXT: */
  h2_2 = __hne2(h2, h2_1);

  // Half Math Functions

  // CHECK: h_2 = sycl::ceil(h);
  h_2 = hceil(h);
  // CHECK: h_2 = sycl::cos(h);
  h_2 = hcos(h);
  // CHECK: h_2 = sycl::exp(h);
  h_2 = hexp(h);
  // CHECK: h_2 = sycl::exp10(h);
  h_2 = hexp10(h);
  // CHECK: h_2 = sycl::exp2(h);
  h_2 = hexp2(h);
  // CHECK: h_2 = sycl::floor(h);
  h_2 = hfloor(h);
  // CHECK: h_2 = sycl::log(h);
  h_2 = hlog(h);
  // CHECK: h_2 = sycl::log10(h);
  h_2 = hlog10(h);
  // CHECK: h_2 = sycl::log2(h);
  h_2 = hlog2(h);
  // CHECK: h_2 = sycl::half_precision::recip<float>(h);
  h_2 = hrcp(h);
  // CHECK: h_2 = sycl::rint(h);
  h_2 = hrint(h);
  // CHECK: h_2 = sycl::rsqrt(h);
  h_2 = hrsqrt(h);
  // CHECK: h_2 = sycl::sin(h);
  h_2 = hsin(h);
  // CHECK: h_2 = sycl::sqrt(h);
  h_2 = hsqrt(h);
  // CHECK: h_2 = sycl::trunc(h);
  h_2 = htrunc(h);

  // Half2 Math Functions

  // CHECK: h2_2 = sycl::ceil(h2);
  h2_2 = h2ceil(h2);
  // CHECK: h2_2 = sycl::cos(h2);
  h2_2 = h2cos(h2);
  // CHECK: h2_2 = sycl::exp(h2);
  h2_2 = h2exp(h2);
  // CHECK: h2_2 = sycl::exp10(h2);
  h2_2 = h2exp10(h2);
  // CHECK: h2_2 = sycl::exp2(h2);
  h2_2 = h2exp2(h2);
  // CHECK: h2_2 = sycl::floor(h2);
  h2_2 = h2floor(h2);
  // CHECK: h2_2 = sycl::log(h2);
  h2_2 = h2log(h2);
  // CHECK: h2_2 = sycl::log10(h2);
  h2_2 = h2log10(h2);
  // CHECK: h2_2 = sycl::log2(h2);
  h2_2 = h2log2(h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of h2rcp is not supported.
  // CHECK-NEXT: */
  h2_2 = h2rcp(h2);
  // CHECK: h2_2 = sycl::rint(h2);
  h2_2 = h2rint(h2);
  // CHECK: h2_2 = sycl::rsqrt(h2);
  h2_2 = h2rsqrt(h2);
  // CHECK: h2_2 = sycl::sin(h2);
  h2_2 = h2sin(h2);
  // CHECK: h2_2 = sycl::sqrt(h2);
  h2_2 = h2sqrt(h2);
  // CHECK: h2_2 = sycl::trunc(h2);
  h2_2 = h2trunc(h2);
}

__global__ void kernelFuncDouble(double *deviceArrayDouble) {
  double &d0 = *deviceArrayDouble, &d1 = *(deviceArrayDouble + 1), &d2 = *(deviceArrayDouble + 2);
  int i;

  // Double Precision Mathematical Functions

  // CHECK: d2 = sycl::acos(d0);
  d2 = acos(d0);
  // CHECK: d2 = sycl::acos((double)i);
  d2 = acos(i);

  // CHECK: d2 = sycl::acosh(d0);
  d2 = acosh(d0);
  // CHECK: d2 = sycl::acosh((double)i);
  d2 = acosh(i);

  // CHECK: d2 = sycl::asin(d0);
  d2 = asin(d0);
  // CHECK: d2 = sycl::asin((double)i);
  d2 = asin(i);

  // CHECK: d2 = sycl::asinh(d0);
  d2 = asinh(d0);
  // CHECK: d2 = sycl::asinh((double)i);
  d2 = asinh(i);

  // CHECK: d2 = sycl::atan2(d0, d1);
  d2 = atan2(d0, d1);
  // CHECK: d2 = sycl::atan2((double)i, (double)i);
  d2 = atan2(i, i);
  // CHECK: d2 = sycl::atan2(d0, (double)i);
  d2 = atan2(d0, i);
  // CHECK: d2 = sycl::atan2((double)i, d1);
  d2 = atan2(i, d1);

  // CHECK: d2 = sycl::atan(d0);
  d2 = atan(d0);
  // CHECK: d2 = sycl::atan((double)i);
  d2 = atan(i);

  // CHECK: d2 = sycl::atanh(d0);
  d2 = atanh(d0);
  // CHECK: d2 = sycl::atanh((double)i);
  d2 = atanh(i);

  // CHECK: d2 = sycl::cbrt(d0);
  d2 = cbrt(d0);
  // CHECK: d2 = sycl::cbrt((double)i);
  d2 = cbrt(i);

  // CHECK: d2 = sycl::ceil(d0);
  d2 = ceil(d0);
  // CHECK: d2 = sycl::ceil((double)i);
  d2 = ceil(i);

  // CHECK: d2 = sycl::copysign(d0, d1);
  d2 = copysign(d0, d1);
  // CHECK: d2 = sycl::copysign((double)i, (double)i);
  d2 = copysign(i, i);
  // CHECK: d2 = sycl::copysign(d0, (double)i);
  d2 = copysign(d0, i);
  // CHECK: d2 = sycl::copysign((double)i, d1);
  d2 = copysign(i, d1);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::cos(d0);
  d2 = cos(d0);
  // CHECK: d2 = sycl::cos((double)i);
  d2 = cos(i);

  // CHECK: d2 = sycl::cosh(d0);
  d2 = cosh(d0);
  // CHECK: d2 = sycl::cosh((double)i);
  d2 = cosh(i);

  // CHECK: d2 = sycl::cospi(d0);
  d2 = cospi(d0);
  // CHECK: d2 = sycl::cospi((double)i);
  d2 = cospi((double)i);

  // CHECK: d2 = sycl::erfc(d0);
  d2 = erfc(d0);
  // CHECK: d2 = sycl::erfc((double)i);
  d2 = erfc(i);

  // CHECK: d2 = sycl::erf(d0);
  d2 = erf(d0);
  // CHECK: d2 = sycl::erf((double)i);
  d2 = erf(i);

  // CHECK: d2 = sycl::exp10(d0);
  d2 = exp10(d0);
  // CHECK: d2 = sycl::exp10((double)i);
  d2 = exp10((double)i);

  // CHECK: d2 = sycl::exp2(d0);
  d2 = exp2(d0);
  // CHECK: d2 = sycl::exp2((double)i);
  d2 = exp2(i);

  // CHECK: d2 = sycl::exp(d0);
  d2 = exp(d0);
  // CHECK: d2 = sycl::exp((double)i);
  d2 = exp(i);

  // CHECK: d2 = sycl::expm1(d0);
  d2 = expm1(d0);
  // CHECK: d2 = sycl::expm1((double)i);
  d2 = expm1(i);

  // CHECK: d2 = sycl::fabs(d0);
  d2 = fabs(d0);
  // CHECK: d2 = sycl::fabs((double)i);
  d2 = fabs(i);

  // CHECK: sycl::fabs(d0);
  abs(d0);
  // CHECK: sycl::fabs(d0 * d1);
  abs(d0 * d1);

  // CHECK: d2 = sycl::fdim(d0, d1);
  d2 = fdim(d0, d1);
  // CHECK: d2 = sycl::fdim((double)i, (double)i);
  d2 = fdim(i, i);
  // CHECK: d2 = sycl::fdim(d0, (double)i);
  d2 = fdim(d0, i);
  // CHECK: d2 = sycl::fdim((double)i, d1);
  d2 = fdim(i, d1);

  // CHECK: d2 = sycl::floor(d0);
  d2 = floor(d0);
  // CHECK: d2 = sycl::floor((double)i);
  d2 = floor(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::fmax(d0, d1);
  d2 = fmax(d0, d1);
  // CHECK: d2 = sycl::fmax((double)i, (double)i);
  d2 = fmax(i, i);
  // CHECK: d2 = sycl::fmax(d0, (double)i);
  d2 = fmax(d0, i);
  // CHECK: d2 = sycl::fmax((double)i, d1);
  d2 = fmax(i, d1);

  // CHECK: d2 = sycl::fmin(d0, d1);
  d2 = fmin(d0, d1);
  // CHECK: d2 = sycl::fmin((double)i, (double)i);
  d2 = fmin(i, i);
  // CHECK: d2 = sycl::fmin(d0, (double)i);
  d2 = fmin(d0, i);
  // CHECK: d2 = sycl::fmin((double)i, d1);
  d2 = fmin(i, d1);

  // CHECK: d2 = sycl::fmod(d0, d1);
  d2 = fmod(d0, d1);
  // CHECK: d2 = sycl::fmod((double)i, (double)i);
  d2 = fmod(i, i);
  // CHECK: d2 = sycl::fmod(d0, (double)i);
  d2 = fmod(d0, i);
  // CHECK: d2 = sycl::fmod((double)i, d1);
  d2 = fmod(i, d1);

  // CHECK: d2 = sycl::frexp(d0, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = frexp(d0, &i);
  // CHECK: d2 = sycl::frexp((double)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = frexp(i, &i);

  // CHECK: d2 = sycl::hypot(d0, d1);
  d2 = hypot(d0, d1);
  // CHECK: d2 = sycl::hypot((double)i, (double)i);
  d2 = hypot(i, i);
  // CHECK: d2 = sycl::hypot(d0, (double)i);
  d2 = hypot(d0, i);
  // CHECK: d2 = sycl::hypot((double)i, d1);
  d2 = hypot(i, d1);

  // CHECK: d2 = sycl::ilogb(d0);
  d2 = ilogb(d0);
  // CHECK: d2 = sycl::ilogb((double)i);
  d2 = ilogb(i);

  // CHECK: d2 = sycl::ldexp(d0, i);
  d2 = ldexp(d0, i);
  // CHECK: d2 = sycl::ldexp((double)i, i);
  d2 = ldexp(i, i);

  // CHECK: d2 = sycl::lgamma(d0);
  d2 = lgamma(d0);
  // CHECK: d2 = sycl::lgamma((double)i);
  d2 = lgamma(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = llrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = llrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = llround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = llround(i);

  // CHECK: d2 = sycl::log10(d0);
  d2 = log10(d0);
  // CHECK: d2 = sycl::log10((double)i);
  d2 = log10(i);

  // CHECK: d2 = sycl::log1p(d0);
  d2 = log1p(d0);
  // CHECK: d2 = sycl::log1p((double)i);
  d2 = log1p(i);

  // CHECK: d2 = sycl::log2(d0);
  d2 = log2(d0);
  // CHECK: d2 = sycl::log2((double)i);
  d2 = log2(i);

  // CHECK: d2 = sycl::logb(d0);
  d2 = logb(d0);
  // CHECK: d2 = sycl::logb((double)i);
  d2 = logb(i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = lrint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = lrint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = lround(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = lround(i);

  // CHECK: d2 = sycl::modf(d0, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d1));
  d2 = modf(d0, &d1);
  // CHECK: d2 = sycl::modf((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d1));
  d2 = modf(i, &d1);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("");

  // CHECK: d2 = sycl::pow<double>(d0, d1);
  d2 = pow(d0, d1);
  // CHECK: d2 = sycl::pown((float)i, i);
  d2 = pow(i, i);
  // CHECK: d2 = sycl::pown(d0, i);
  d2 = pow(d0, i);
  // CHECK: d2 = sycl::pow<double>(i, d1);
  d2 = pow(i, d1);

  // CHECK: sycl::pown(f, 1);
  float f;
  pow(f, 1);

  // CHECK: d2 = sycl::remainder(d0, d1);
  d2 = remainder(d0, d1);
  // CHECK: d2 = sycl::remainder((double)i, (double)i);
  d2 = remainder(i, i);
  // CHECK: d2 = sycl::remainder(d0, (double)i);
  d2 = remainder(d0, i);
  // CHECK: d2 = sycl::remainder((double)i, d1);
  d2 = remainder(i, d1);

  // CHECK: d2 = sycl::remquo(d0, d1, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = remquo(d0, d1, &i);
  // CHECK: d2 = sycl::remquo((double)i, (double)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = remquo(i, i, &i);
  // CHECK: d2 = sycl::remquo(d0, (double)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = remquo(d0, i, &i);
  // CHECK: d2 = sycl::remquo((double)i, d1, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  d2 = remquo(i, d1, &i);

  // CHECK: d2 = sycl::rint(d0);
  d2 = rint(d0);
  // CHECK: d2 = sycl::rint((double)i);
  d2 = rint(i);

  // CHECK: d2 = sycl::round(d0);
  d2 = round(d0);
  // CHECK: d2 = sycl::round((double)i);
  d2 = round(i);

  // CHECK: d2 = sycl::rsqrt(d0);
  d2 = rsqrt(d0);
  // CHECK: d2 = sycl::rsqrt((double)i);
  d2 = rsqrt((double)i);

  // CHECK: d1 = sycl::sincos(d0, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d2));
  sincos(d0, &d1, &d2);
  // CHECK: d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d2));
  sincos(i, &d1, &d2);

  // CHECK: d2 = sycl::sin(d0);
  d2 = sin(d0);
  // CHECK: d2 = sycl::sin((double)i);
  d2 = sin(i);

  // CHECK: d2 = sycl::sinh(d0);
  d2 = sinh(d0);
  // CHECK: d2 = sycl::sinh((double)i);
  d2 = sinh(i);

  // CHECK: d2 = sycl::sinpi(d0);
  d2 = sinpi(d0);
  // CHECK: d2 = sycl::sinpi((double)i);
  d2 = sinpi((double)i);

  // CHECK: d2 = sycl::sqrt(d0);
  d2 = sqrt(d0);
  // CHECK: d2 = sycl::sqrt((double)i);
  d2 = sqrt(i);

  // CHECK: d2 = sycl::tan(d0);
  d2 = tan(d0);
  // CHECK: d2 = sycl::tan((double)i);
  d2 = tan(i);

  // CHECK: d2 = sycl::tanh(d0);
  d2 = tanh(d0);
  // CHECK: d2 = sycl::tanh((double)i);
  d2 = tanh(i);

  // CHECK: d2 = sycl::tgamma(d0);
  d2 = tgamma(d0);
  // CHECK: d2 = sycl::tgamma((double)i);
  d2 = tgamma(i);

  // CHECK: d2 = sycl::trunc(d0);
  d2 = trunc(d0);
  // CHECK: d2 = sycl::trunc((double)i);
  d2 = trunc(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 + d1;
  d2 = __dadd_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 / d1;
  d2 = __ddiv_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 * d1;
  d2 = __dmul_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rn(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/d0);
  d1 = __drcp_rz(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = (1.0/(d0+d0));
  d1 = __drcp_rz(d0+d0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rn(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt(d0);
  d0 = __dsqrt_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt(d1);
  d1 = __dsqrt_rz(d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d0 = sycl::sqrt((double)i);
  d0 = __dsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d1 = sycl::sqrt((double)i);
  d1 = __dsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rd(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rn(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_ru(d0, d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = d0 - d1;
  d2 = __dsub_rz(d0, d1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rd(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rn(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_ru(d0, d1, d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma(d0, d1, d2);
  d2 = __fma_rz(d0, d1, d2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = __fma_rz(i, i, i);

  // CHECK: d0 = sycl::fmin(d0, d1);
  d0 = fmin(d0, d1);
  // CHECK: d0 = sycl::fmin((double)i, (double)i);
  d0 = fmin(i, i);
  // CHECK: d0 = sycl::fmin(d0, (double)i);
  d0 = fmin(d0, i);
  // CHECK: d0 = sycl::fmin((double)i, d1);
  d0 = fmin(i, d1);

  // CHECK: d0 = sycl::fmax(d0, d1);
  d0 = fmax(d0, d1);
  // CHECK: d0 = sycl::fmax((double)i, (double)i);
  d0 = fmax(i, i);
  // CHECK: d0 = sycl::fmax(d0, (double)i);
  d0 = fmax(d0, i);
  // CHECK: d0 = sycl::fmax((double)i, d1);
  d0 = fmax(i, d1);

  // CHECK: d1 = sycl::floor(d1);
  d1 = floor(d1);
  // CHECK: d1 = sycl::floor((double)i);
  d1 = floor(i);

  // CHECK: d2 = sycl::ceil(d2);
  d2 = ceil(d2);
  // CHECK: d2 = sycl::ceil((double)i);
  d2 = ceil(i);

  // CHECK: d2 = sycl::fma(d0, d1, d2);
  d2 = fma(d0, d1, d2);
  // CHECK: d2 = sycl::fma((double)i, (double)i, (double)i);
  d2 = fma(i, i, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, (double)i);
  d2 = fma(d0, i, i);
  // CHECK: d2 = sycl::fma((double)i, d1, (double)i);
  d2 = fma(i, d1, i);
  // CHECK: d2 = sycl::fma((double)i, (double)i, d2);
  d2 = fma(i, i, d2);
  // CHECK: d2 = sycl::fma(d0, d1, (double)i);
  d2 = fma(d0, d1, i);
  // CHECK: d2 = sycl::fma(d0, (double)i, d2);
  d2 = fma(d0, i, d2);
  // CHECK: d2 = sycl::fma((double)i, d1, d2);
  d2 = fma(i, d1, d2);

  // CHECK: d2 = sycl::nan(0u);
  d2 = nan("NaN");

  // CHECK: d0 = sycl::nextafter(d0, d0);
  d0 = nextafter(d0, d0);
  // CHECK: d0 = sycl::nextafter((double)i, (double)i);
  d0 = nextafter(i, i);
  // CHECK: d0 = sycl::nextafter(d0, (double)i);
  d0 = nextafter(d0, i);
  // CHECK: d0 = sycl::nextafter((double)i, d1);
  d0 = nextafter(i, d1);
}

__global__ void kernelFuncFloat(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1), &f2 = *(deviceArrayFloat + 2);
  int i;

  // Single Precision Mathematical Functions

  // CHECK: f2 = sycl::log(f0);
  f2 = logf(f0);
  // CHECK: f2 = sycl::log((float)i);
  f2 = logf(i);

  // CHECK: f2 = sycl::acos(f0);
  f2 = acosf(f0);
  // CHECK: f2 = sycl::acos((float)i);
  f2 = acosf(i);

  // CHECK: f2 = sycl::acosh(f0);
  f2 = acoshf(f0);
  // CHECK: f2 = sycl::acosh((float)i);
  f2 = acoshf(i);

  // CHECK: f2 = sycl::asin(f0);
  f2 = asinf(f0);
  // CHECK: f2 = sycl::asin((float)i);
  f2 = asinf(i);

  // CHECK: f2 = sycl::asinh(f0);
  f2 = asinhf(f0);
  // CHECK: f2 = sycl::asinh((float)i);
  f2 = asinhf(i);

  // CHECK: f2 = sycl::atan2(f0, f1);
  f2 = atan2f(f0, f1);
  // CHECK: f2 = sycl::atan2((float)i, (float)i);
  f2 = atan2f(i, i);
  // CHECK: f2 = sycl::atan2(f0, (float)i);
  f2 = atan2f(f0, i);
  // CHECK: f2 = sycl::atan2((float)i, f1);
  f2 = atan2f(i, f1);

  // CHECK: f2 = sycl::atan(f0);
  f2 = atanf(f0);
  // CHECK: f2 = sycl::atan((float)i);
  f2 = atanf(i);

  // CHECK: f2 = sycl::atanh(f0);
  f2 = atanhf(f0);
  // CHECK: f2 = sycl::atanh((float)i);
  f2 = atanhf(i);

  // CHECK: f2 = sycl::cbrt(f0);
  f2 = cbrtf(f0);
  // CHECK: f2 = sycl::cbrt((float)i);
  f2 = cbrtf(i);

  // CHECK: f2 = sycl::ceil(f0);
  f2 = ceilf(f0);
  // CHECK: f2 = sycl::ceil((float)i);
  f2 = ceilf(i);

  // CHECK: f2 = sycl::copysign(f0, f1);
  f2 = copysignf(f0, f1);
  // CHECK: f2 = sycl::copysign((float)i, (float)i);
  f2 = copysignf(i, i);
  // CHECK: f2 = sycl::copysign(f0, (float)i);
  f2 = copysignf(f0, i);
  // CHECK: f2 = sycl::copysign((float)i, f1);
  f2 = copysignf(i, f1);

  // CHECK: f2 = sycl::cos(f0);
  f2 = cosf(f0);
  // CHECK: f2 = sycl::cos((float)i);
  f2 = cosf(i);

  // CHECK: f2 = sycl::cosh(f0);
  f2 = coshf(f0);
  // CHECK: f2 = sycl::cosh((float)i);
  f2 = coshf(i);

  // CHECK: f2 = sycl::cospi(f0);
  f2 = cospif(f0);
  // CHECK: f2 = sycl::cospi((float)i);
  f2 = cospif(i);

  // CHECK: f2 = sycl::erfc(f0);
  f2 = erfcf(f0);
  // CHECK: f2 = sycl::erfc((float)i);
  f2 = erfcf(i);

  // CHECK: f2 = sycl::erf(f0);
  f2 = erff(f0);
  // CHECK: f2 = sycl::erf((float)i);
  f2 = erff(i);

  // CHECK: f2 = sycl::exp10(f0);
  f2 = exp10f(f0);
  // CHECK: f2 = sycl::exp10((float)i);
  f2 = exp10f(i);

  // CHECK: f2 = sycl::exp2(f0);
  f2 = exp2f(f0);
  // CHECK: f2 = sycl::exp2((float)i);
  f2 = exp2f(i);

  // CHECK: f2 = sycl::exp(f0);
  f2 = expf(f0);
  // CHECK: f2 = sycl::exp((float)i);
  f2 = expf(i);

  // CHECK: f2 = sycl::expm1(f0);
  f2 = expm1f(f0);
  // CHECK: f2 = sycl::expm1((float)i);
  f2 = expm1f(i);

  // CHECK: f2 = sycl::fabs(f0);
  f2 = fabsf(f0);
  // CHECK: f2 = sycl::fabs((float)i);
  f2 = fabsf(i);

  // CHECK: f2 = sycl::fdim(f0, f1);
  f2 = fdimf(f0, f1);
  // CHECK: f2 = sycl::fdim((float)i, (float)i);
  f2 = fdimf(i, i);
  // CHECK: f2 = sycl::fdim(f0, (float)i);
  f2 = fdimf(f0, i);
  // CHECK: f2 = sycl::fdim((float)i, f1);
  f2 = fdimf(i, f1);

  // CHECK: f2 = f0 / f1;
  f2 = fdividef(f0, f1);
  // CHECK: f2 = i / i;
  f2 = fdividef(i, i);
  // CHECK: f2 = f0 / i;
  f2 = fdividef(f0, i);
  // CHECK: f2 = i / f1;
  f2 = fdividef(i, f1);

  // CHECK: f2 = sycl::floor(f0);
  f2 = floorf(f0);
  // CHECK: f2 = sycl::floor((float)i);
  f2 = floorf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f2 = sycl::fmin(f0, f1);
  f2 = fminf(f0, f1);
  // CHECK: f2 = sycl::fmin((float)i, (float)i);
  f2 = fminf(i, i);
  // CHECK: f2 = sycl::fmin(f0, (float)i);
  f2 = fminf(f0, i);
  // CHECK: f2 = sycl::fmin((float)i, f1);
  f2 = fminf(i, f1);

  // CHECK: f2 = sycl::fmod(f0, f1);
  f2 = fmodf(f0, f1);
  // CHECK: f2 = sycl::fmod((float)i, (float)i);
  f2 = fmodf(i, i);
  // CHECK: f2 = sycl::fmod(f0, (float)i);
  f2 = fmodf(f0, i);
  // CHECK: f2 = sycl::fmod((float)i, f1);
  f2 = fmodf(i, f1);

  // CHECK: f2 = sycl::frexp(f0, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = frexpf(f0, &i);
  // CHECK: f2 = sycl::frexp((float)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = frexpf(i, &i);

  // CHECK: f2 = sycl::hypot(f0, f1);
  f2 = hypotf(f0, f1);
  // CHECK: f2 = sycl::hypot((float)i, (float)i);
  f2 = hypotf(i, i);
  // CHECK: f2 = sycl::hypot(f0, (float)i);
  f2 = hypotf(f0, i);
  // CHECK: f2 = sycl::hypot((float)i, f1);
  f2 = hypotf(i, f1);

  // CHECK: f2 = sycl::ilogb(f0);
  f2 = ilogbf(f0);
  // CHECK: f2 = sycl::ilogb((float)i);
  f2 = ilogbf(i);

  // CHECK: i = sycl::isfinite(f0);
  i = isfinite(f0);
  // CHECK: i = sycl::isfinite((float)i);
  i = isfinite(i);

  // CHECK: i = sycl::isinf(f0);
  i = isinf(f0);
  // CHECK: i = sycl::isinf((float)i);
  i = isinf(i);

  // CHECK: i = sycl::isnan(f0);
  i = isnan(f0);
  // CHECK: i = sycl::isnan((float)i);
  i = isnan(i);

  // CHECK: f2 = sycl::ldexp(f0, i);
  f2 = ldexpf(f0, i);
  // CHECK: f2 = sycl::ldexp((float)i, i);
  f2 = ldexpf(i, i);

  // CHECK: f2 = sycl::lgamma(f0);
  f2 = lgammaf(f0);
  // CHECK: f2 = sycl::lgamma((float)i);
  f2 = lgammaf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = llrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = llrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = llroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = llroundf(i);

  // CHECK: f2 = sycl::log10(f0);
  f2 = log10f(f0);
  // CHECK: f2 = sycl::log10((float)i);
  f2 = log10f(i);

  // CHECK: f2 = sycl::log1p(f0);
  f2 = log1pf(f0);
  // CHECK: f2 = sycl::log1p((float)i);
  f2 = log1pf(i);

  // CHECK: f2 = sycl::log2(f0);
  f2 = log2f(f0);
  // CHECK: f2 = sycl::log2((float)i);
  f2 = log2f(i);

  // CHECK: f2 = sycl::logb(f0);
  f2 = logbf(f0);
  // CHECK: f2 = sycl::logb((float)i);
  f2 = logbf(i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = lrintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = lrintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = lroundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = lroundf(i);

  // CHECK: f2 = sycl::modf(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f1));
  f2 = modff(f0, &f1);
  // CHECK: f2 = sycl::modf((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f1));
  f2 = modff(i, &f1);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nan("");

  // CHECK: f2 = sycl::pow<float>(f0, f1);
  f2 = powf(f0, f1);
  // CHECK: f2 = sycl::pown((float)i, i);
  f2 = powf(i, i);
  // CHECK: f2 = sycl::pown(f0, i);
  f2 = powf(f0, i);
  // CHECK: f2 = sycl::pow<float>(i, f1);
  f2 = powf(i, f1);

  // CHECK: f2 = sycl::remainder(f0, f1);
  f2 = remainderf(f0, f1);
  // CHECK: f2 = sycl::remainder((float)i, (float)i);
  f2 = remainderf(i, i);
  // CHECK: f2 = sycl::remainder(f0, (float)i);
  f2 = remainderf(f0, i);
  // CHECK: f2 = sycl::remainder((float)i, f1);
  f2 = remainderf(i, f1);

  // CHECK: f2 = sycl::remquo(f0, f1, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = remquof(f0, f1, &i);
  // CHECK: f2 = sycl::remquo((float)i, (float)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = remquof(i, i, &i);
  // CHECK: f2 = sycl::remquo(f0, (float)i, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = remquof(f0, i, &i);
  // CHECK: f2 = sycl::remquo((float)i, f1, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  f2 = remquof(i, f1, &i);

  // CHECK: f2 = sycl::rint(f0);
  f2 = rintf(f0);
  // CHECK: f2 = sycl::rint((float)i);
  f2 = rintf(i);

  // CHECK: f2 = sycl::round(f0);
  f2 = roundf(f0);
  // CHECK: f2 = sycl::round((float)i);
  f2 = roundf(i);

  // CHECK: f2 = sycl::rsqrt(f0);
  f2 = rsqrtf(f0);
  // CHECK: f2 = sycl::rsqrt((float)i);
  f2 = rsqrtf(i);

  // CHECK: f2 = sycl::signbit(f0);
  f2 = signbit(f0);
  // CHECK: f2 = sycl::signbit((float)i);
  f2 = signbit(i);

  // CHECK: f1 = sycl::sincos(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  sincosf(i, &f1, &f2);

  // CHECK: f2 = sycl::sin(f0);
  f2 = sinf(f0);
  // CHECK: f2 = sycl::sin((float)i);
  f2 = sinf(i);

  // CHECK: f2 = sycl::sinh(f0);
  f2 = sinhf(f0);
  // CHECK: f2 = sycl::sinh((float)i);
  f2 = sinhf(i);

  // CHECK: f2 = sycl::sinpi(f0);
  f2 = sinpif(f0);
  // CHECK: f2 = sycl::sinpi((float)i);
  f2 = sinpif(i);

  // CHECK: f2 = sycl::sqrt(f0);
  f2 = sqrtf(f0);
  // CHECK: f2 = sycl::sqrt((float)i);
  f2 = sqrtf(i);

  // CHECK: f2 = sycl::tan(f0);
  f2 = tanf(f0);
  // CHECK: f2 = sycl::tan((float)i);
  f2 = tanf(i);

  // CHECK: f2 = sycl::tanh(f0);
  f2 = tanhf(f0);
  // CHECK: f2 = sycl::tanh((float)i);
  f2 = tanhf(i);

  // CHECK: f2 = sycl::tgamma(f0);
  f2 = tgammaf(f0);
  // CHECK: f2 = sycl::tgamma((float)i);
  f2 = tgammaf(i);

  // CHECK: f2 = sycl::trunc(f0);
  f2 = truncf(f0);
  // CHECK: f2 = sycl::trunc((float)i);
  f2 = truncf(i);

  // CHECK: f0 = sycl::cos(f0);
  f0 = __cosf(f0);
  // CHECK: f0 = sycl::cos((float)i);
  f0 = __cosf(i);

  // CHECK: f0 = sycl::exp10(f0);
  f0 = __exp10f(f0);
  // CHECK: f0 = sycl::exp10((float)i);
  f0 = __exp10f(i);

  // CHECK: f0 = sycl::exp(f0);
  f0 = __expf(f0);
  // CHECK: f0 = sycl::exp((float)i);
  f0 = __expf(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 + f1;
  f2 = __fadd_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 / f1;
  f2 = __fdiv_rz(f0, f1);

  // CHECK: f2 = f0 / f1;
  f2 = __fdividef(f0, f1);
  // CHECK: f2 = i / i;
  f2 = __fdividef(i, i);
  // CHECK: f2 = f0 / i;
  f2 = __fdividef(f0, i);
  // CHECK: f2 = i / f1;
  f2 = __fdividef(i, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rd(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rn(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_ru(f0, f1, f2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma(f0, f1, f2);
  f2 = __fmaf_rz(f0, f1, f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rd(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rn(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_ru(i, i, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = __fmaf_rz(i, i, i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK: f2 = f0 * f1;
  f2 = __fmul_rz(f0, f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rn(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip(f0);
  f1 = __frcp_rz(f0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::native::recip((float)i);
  f1 = __frcp_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_rd(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rn(f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt(f0);
  f0 = __fsqrt_ru(f0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt(f1);
  f1 = __fsqrt_rz(f1);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_rd(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rn(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f0 = sycl::sqrt((float)i);
  f0 = __fsqrt_ru(i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f1 = sycl::sqrt((float)i);
  f1 = __fsqrt_rz(i);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rd(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rn(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_ru(f0, f1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = f0 - f1;
  f2 = __fsub_rz(f0, f1);

  // CHECK: f1 = sycl::log10(f1);
  f1 = __log10f(f1);
  // CHECK: f1 = sycl::log10((float)i);
  f1 = __log10f(i);

  // CHECK: f1 = sycl::log2(f1);
  f1 = __log2f(f1);
  // CHECK: f1 = sycl::log2((float)i);
  f1 = __log2f(i);

  // CHECK: f1 = sycl::log(f1);
  f1 = __logf(f1);
  // CHECK: f1 = sycl::log((float)i);
  f1 = __logf(i);

  // CHECK: f2 = sycl::pow<float>(f0, f1);
  f2 = __powf(f0, f1);
  // CHECK: f2 = sycl::pown((float)i, i);
  f2 = __powf(i, i);
  // CHECK: f2 = sycl::pown(f0, i);
  f2 = __powf(f0, i);
  // CHECK: f2 = sycl::pow<float>(i, f1);
  f2 = __powf(i, f1);

  // CHECK: f1 = sycl::sincos(f0, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  __sincosf(f0, &f1, &f2);
  // CHECK: f1 = sycl::sincos((float)i, sycl::make_ptr<float, sycl::access::address_space::global_space>(&f2));
  __sincosf(i, &f1, &f2);

  // CHECK: f1 = sycl::sin(f1);
  f1 = __sinf(f1);
  // CHECK: f1 = sycl::sin((float)i);
  f1 = __sinf(i);

  // CHECK: f1 = sycl::tan(f1);
  f1 = __tanf(f1);
  // CHECK: f1 = sycl::tan((float)i);
  f1 = __tanf(i);

  // CHECK: f0 = sycl::fmin(f0, f1);
  f0 = fminf(f0, f1);
  // CHECK: f0 = sycl::fmin((float)i, (float)i);
  f0 = fminf(i, i);
  // CHECK: f0 = sycl::fmin(f0, (float)i);
  f0 = fminf(f0, i);
  // CHECK: f0 = sycl::fmin((float)i, f1);
  f0 = fminf(i, f1);

  // CHECK: f2 = sycl::fmax(f0, f1);
  f2 = fmaxf(f0, f1);
  // CHECK: f2 = sycl::fmax((float)i, (float)i);
  f2 = fmaxf(i, i);
  // CHECK: f2 = sycl::fmax(f0, (float)i);
  f2 = fmaxf(f0, i);
  // CHECK: f2 = sycl::fmax((float)i, f1);
  f2 = fmaxf(i, f1);

  // CHECK: f1 = sycl::floor(f1);
  f1 = floorf(f1);
  // CHECK: f1 = sycl::floor((float)i);
  f1 = floorf(i);

  // CHECK: f2 = sycl::ceil(f2);
  f2 = ceilf(f2);
  // CHECK: f2 = sycl::ceil((float)i);
  f2 = ceilf(i);

  // CHECK: f2 = sycl::fma(f0, f1, f2);
  f2 = fmaf(f0, f1, f2);
  // CHECK: f2 = sycl::fma((float)i, (float)i, (float)i);
  f2 = fmaf(i, i, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, (float)i);
  f2 = fmaf(f0, i, i);
  // CHECK: f2 = sycl::fma((float)i, f1, (float)i);
  f2 = fmaf(i, f1, i);
  // CHECK: f2 = sycl::fma((float)i, (float)i, f2);
  f2 = fmaf(i, i, f2);
  // CHECK: f2 = sycl::fma(f0, f1, (float)i);
  f2 = fmaf(f0, f1, i);
  // CHECK: f2 = sycl::fma(f0, (float)i, f2);
  f2 = fmaf(f0, i, f2);
  // CHECK: f2 = sycl::fma((float)i, f1, f2);
  f2 = fmaf(i, f1, f2);

  // CHECK: f2 = sycl::nan(0u);
  f2 = nanf("NaN");

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt(f2);
  f2 = __frsqrt_rn(f2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: f2 = sycl::rsqrt((float)i);
  f2 = __frsqrt_rn(i);

  // CHECK: f0 = sycl::nextafter(f0, f0);
  f0 = nextafterf(f0, f0);
  // CHECK: f0 = sycl::nextafter((float)i, (float)i);
  f0 = nextafterf(i, i);
  // CHECK: f0 = sycl::nextafter(f0, (float)i);
  f0 = nextafterf(f0, i);
  // CHECK: f0 = sycl::nextafter((float)i, f1);
  f0 = nextafterf(i, f1);
}

__global__ void kernelFuncTypecasts() {
  short s, s_1;
  unsigned short us;
  int i, i_1;
  unsigned int ui, ui_1;
  long l;
  unsigned long ul;
  long long ll;
  unsigned long long ull;

  __half h;
  __half2 h2;
  float f;
  float2 f2;
  double d;
  double2 d2;

  // CHECK: h2 = f2.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float22half2_rn(f2);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::automatic>()[0];
  h = __float2half(f);

  // CHECK: h2 = sycl::float2{f,f}.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __float2half2_rn(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __float2half_rd(f);

  // sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  __float2half_rn(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __float2half_ru(f);

  // CHECK: h = sycl::vec<float, 1>{f}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __float2half_rz(f);

  // CHECK: h2 = sycl::float2{f,f}.convert<sycl::half, sycl::rounding_mode::rte>();
  h2 = __floats2half2_rn(f, f);

  // CHECK: f2 = h2.convert<float, sycl::rounding_mode::automatic>();
  f2 = __half22float2(h2);

  // CHECK: f = sycl::vec<sycl::half, 1>{h}.convert<float, sycl::rounding_mode::automatic>()[0];
  f = __half2float(h);

  // CHECK: h2 = sycl::half2{h,h};
  h2 = __half2half2(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtn>()[0];
  i = __half2int_rd(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rte>()[0];
  i = __half2int_rn(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtp>()[0];
  i = __half2int_ru(h);

  // CHECK: i = sycl::vec<sycl::half, 1>{h}.convert<int, sycl::rounding_mode::rtz>()[0];
  i = __half2int_rz(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __half2ll_rd(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __half2ll_rn(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __half2ll_ru(h);

  // CHECK: ll = sycl::vec<sycl::half, 1>{h}.convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __half2ll_rz(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtn>()[0];
  s = __half2short_rd(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rte>()[0];
  s = __half2short_rn(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtp>()[0];
  s = __half2short_ru(h);

  // CHECK: s = sycl::vec<sycl::half, 1>{h}.convert<short, sycl::rounding_mode::rtz>()[0];
  s = __half2short_rz(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtn>()[0];
  ui = __half2uint_rd(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rte>()[0];
  ui = __half2uint_rn(h);

  // CHECK:ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtp>()[0];
  ui = __half2uint_ru(h);

  // CHECK: ui = sycl::vec<sycl::half, 1>{h}.convert<unsigned int, sycl::rounding_mode::rtz>()[0];
  ui = __half2uint_rz(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __half2ull_rd(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __half2ull_rn(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __half2ull_ru(h);

  // CHECK: ull = sycl::vec<sycl::half, 1>{h}.convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __half2ull_rz(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtn>()[0];
  us = __half2ushort_rd(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rte>()[0];
  us = __half2ushort_rn(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtp>()[0];
  us = __half2ushort_ru(h);

  // CHECK: us = sycl::vec<sycl::half, 1>{h}.convert<unsigned short, sycl::rounding_mode::rtz>()[0];
  us = __half2ushort_rz(h);

  // CHECK: s = sycl::bit_cast<short>(h);
  s = __half_as_short(h);

  // CHECK: us = sycl::bit_cast<unsigned short>(h);
  us = __half_as_ushort(h);

  // CHECK: h2 = sycl::half2{h,h};
  h2 = __halves2half2(h, h);

  // CHECK: f = h2[0];
  f = __high2float(h2);

  // CHECK: h = h2[0];
  h = __high2half(h2);

  // CHECK: h2 = sycl::half2{h2[0], h2[0]};
  h2 = __high2half2(h2);

  // CHECK: h2 = sycl::half2{h2[0], h2[0]};
  h2 = __highs2half2(h2, h2);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __int2half_rd(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __int2half_rn(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __int2half_ru(i);

  // CHECK: h = sycl::vec<int, 1>{i}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __int2half_rz(i);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ll2half_rd(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ll2half_rn(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ll2half_ru(ll);

  // CHECK: h = sycl::vec<long long, 1>{ll}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ll2half_rz(ll);

  // CHECK: f = h2[1];
  f = __low2float(h2);

  // CHECK: h = h2[1];
  h = __low2half(h2);

  // CHECK: h2 = sycl::half2{h2[1], h2[1]};
  h2 = __low2half2(h2);

  // CHECK: h2 = sycl::half2{h2[1], h2[0]};
  h2 = __lowhigh2highlow(h2);

  // CHECK: h2 = sycl::half2{h2[1], h2[1]};
  h2 = __lows2half2(h2, h2);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __short2half_rd(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __short2half_rn(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __short2half_ru(s);

  // CHECK: h = sycl::vec<short, 1>{s}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __short2half_rz(s);

  // CHECK: h = sycl::bit_cast<sycl::half>(s);
  h = __short_as_half(s);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __uint2half_rd(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __uint2half_rn(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __uint2half_ru(ui);

  // CHECK: h = sycl::vec<unsigned int, 1>{ui}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __uint2half_rz(ui);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ull2half_rd(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ull2half_rn(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ull2half_ru(ull);

  // CHECK: h = sycl::vec<unsigned long long, 1>{ull}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ull2half_rz(ull);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtn>()[0];
  h = __ushort2half_rd(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rte>()[0];
  h = __ushort2half_rn(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtp>()[0];
  h = __ushort2half_ru(us);

  // CHECK: h = sycl::vec<unsigned short, 1>{us}.convert<sycl::half, sycl::rounding_mode::rtz>()[0];
  h = __ushort2half_rz(us);

  // CHECK: h = sycl::bit_cast<sycl::half>(us);
  h = __ushort_as_half(us);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __double2float_rd(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __double2float_rn(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __double2float_ru(d);

  // CHECK: f = sycl::vec<double, 1>{d}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __double2float_rz(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtn>()[0];
  i = __double2int_rd(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rte>()[0];
  i = __double2int_rn(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtp>()[0];
  i = __double2int_ru(d);

  // CHECK: i = sycl::vec<double, 1>{d}.convert<int, sycl::rounding_mode::rtz>()[0];
  i = __double2int_rz(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __double2ll_rd(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __double2ll_rn(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __double2ll_ru(d);

  // CHECK: ll = sycl::vec<double, 1>{d}.convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __double2ll_rz(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtn>()[0];
  ui = __double2uint_rd(d);

  // CHECK:ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rte>()[0];
  ui = __double2uint_rn(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtp>()[0];
  ui = __double2uint_ru(d);

  // CHECK: ui = sycl::vec<double, 1>{d}.convert<unsigned int, sycl::rounding_mode::rtz>()[0];
  ui = __double2uint_rz(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __double2ull_rd(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __double2ull_rn(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __double2ull_ru(d);

  // CHECK: ull = sycl::vec<double, 1>{d}.convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __double2ull_rz(d);

  // CHECK: ll = sycl::bit_cast<long long>(d);
  ll = __double_as_longlong(d);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtn>()[0];
  i = __float2int_rd(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rte>()[0];
  i = __float2int_rn(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtp>()[0];
  i = __float2int_ru(f);

  // CHECK: i = sycl::vec<float, 1>{f}.convert<int, sycl::rounding_mode::rtz>()[0];
  i = __float2int_rz(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtn>()[0];
  ll = __float2ll_rd(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rte>()[0];
  ll = __float2ll_rn(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtp>()[0];
  ll = __float2ll_ru(f);

  // CHECK: ll = sycl::vec<float, 1>{f}.convert<long long, sycl::rounding_mode::rtz>()[0];
  ll = __float2ll_rz(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtn>()[0];
  ui = __float2uint_rd(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rte>()[0];
  ui = __float2uint_rn(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtp>()[0];
  ui = __float2uint_ru(f);

  // CHECK: ui = sycl::vec<float, 1>{f}.convert<unsigned int, sycl::rounding_mode::rtz>()[0];
  ui = __float2uint_rz(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtn>()[0];
  ull = __float2ull_rd(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rte>()[0];
  ull = __float2ull_rn(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtp>()[0];
  ull = __float2ull_ru(f);

  // CHECK: ull = sycl::vec<float, 1>{f}.convert<unsigned long long, sycl::rounding_mode::rtz>()[0];
  ull = __float2ull_rz(f);

  // CHECK: i = sycl::bit_cast<int>(f);
  i = __float_as_int(f);

  // CHECK: ui = sycl::bit_cast<unsigned int>(f);
  ui = __float_as_uint(f);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __int2double_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtn>()[0];
  d = __int2float_rd(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rte>()[0];
  d = __int2float_rn(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtp>()[0];
  d = __int2float_ru(i);

  // CHECK: d = sycl::vec<int, 1>{i}.convert<float, sycl::rounding_mode::rtz>()[0];
  d = __int2float_rz(i);

  // CHECK: f = sycl::bit_cast<float>(i);
  f = __int_as_float(i);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtn>()[0];
  d = __ll2double_rd(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __ll2double_rn(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtp>()[0];
  d = __ll2double_ru(ll);

  // CHECK: d = sycl::vec<long long, 1>{ll}.convert<double, sycl::rounding_mode::rtz>()[0];
  d = __ll2double_rz(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __ll2float_rd(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __ll2float_rn(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __ll2float_ru(ll);

  // CHECK: f = sycl::vec<long long, 1>{ll}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __ll2float_rz(ll);

  // CHECK: d = sycl::bit_cast<double>(ll);
  d = __longlong_as_double(ll);

  // CHECK: d = sycl::vec<unsigned int, 1>{ui}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __uint2double_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __uint2float_rd(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __uint2float_rn(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __uint2float_ru(ui);

  // CHECK: f = sycl::vec<unsigned int, 1>{ui}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __uint2float_rz(ui);

  // CHECK: f = sycl::bit_cast<float>(ui);
  f = __uint_as_float(ui);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtn>()[0];
  d = __ull2double_rd(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rte>()[0];
  d = __ull2double_rn(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtp>()[0];
  d = __ull2double_ru(ull);

  // CHECK: d = sycl::vec<unsigned long long, 1>{ull}.convert<double, sycl::rounding_mode::rtz>()[0];
  d = __ull2double_rz(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtn>()[0];
  f = __ull2float_rd(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rte>()[0];
  f = __ull2float_rn(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtp>()[0];
  f = __ull2float_ru(ull);

  // CHECK: f = sycl::vec<unsigned long long, 1>{ull}.convert<float, sycl::rounding_mode::rtz>()[0];
  f = __ull2float_rz(ull);
}

void testDouble() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(double);

  double *hostArrayDouble = (double *)malloc(bytes);
  memset(hostArrayDouble, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayDouble = pi;
  *(hostArrayDouble + 1) = pi - 1;

  double *deviceArrayDouble;
  hipMalloc((double **)&deviceArrayDouble, bytes);

  hipMemcpy(deviceArrayDouble, hostArrayDouble, bytes, hipMemcpyHostToDevice);

  kernelFuncDouble<<<1, 1>>>(deviceArrayDouble);

  hipMemcpy(hostArrayDouble, deviceArrayDouble, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayDouble);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayDouble + 2) << endl;
}

void testFloat() {
  const unsigned int NUM = 3;
  const unsigned int bytes = NUM * sizeof(float);

  float *hostArrayFloat = (float *)malloc(bytes);
  memset(hostArrayFloat, 0, bytes);
  const long double pi = std::acos(-1.L);
  *hostArrayFloat = pi;
  *(hostArrayFloat + 1) = pi - 1;

  float *deviceArrayFloat;
  hipMalloc((float **)&deviceArrayFloat, bytes);

  hipMemcpy(deviceArrayFloat, hostArrayFloat, bytes, hipMemcpyHostToDevice);

  kernelFuncFloat<<<1, 1>>>(deviceArrayFloat);

  hipMemcpy(hostArrayFloat, deviceArrayFloat, bytes, hipMemcpyDeviceToHost);

  hipFree(deviceArrayFloat);

  cout << std::setprecision(std::numeric_limits<long double>::digits10 + 1)
       << *(hostArrayFloat + 2) << endl;
}

__global__ void testUnsupported() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;
  half h;
  float f;
  double d;
  half2 h2;
  bool b;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hadd_sat is not supported.
  // CHECK-NEXT: */
  __hadd_sat(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma_sat is not supported.
  // CHECK-NEXT: */
  __hfma_sat(h, h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hmul_sat is not supported.
  // CHECK-NEXT: */
  __hmul_sat(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hsub_sat is not supported.
  // CHECK-NEXT: */
  __hsub_sat(h, h);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hadd2_sat is not supported.
  // CHECK-NEXT: */
  __hadd2_sat(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hfma2_sat is not supported.
  // CHECK-NEXT: */
  __hfma2_sat(h2, h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hmul2_sat is not supported.
  // CHECK-NEXT: */
  __hmul2_sat(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hsub2_sat is not supported.
  // CHECK-NEXT: */
  __hsub2_sat(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hequ is not supported.
  // CHECK-NEXT: */
  b = __hequ(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hgeu is not supported.
  // CHECK-NEXT: */
  b = __hgeu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hgtu is not supported.
  // CHECK-NEXT: */
  b = __hgtu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hleu is not supported.
  // CHECK-NEXT: */
  b = __hleu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hltu is not supported.
  // CHECK-NEXT: */
  b = __hltu(h, h);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hneu is not supported.
  // CHECK-NEXT: */
  b = __hneu(h, h);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbeq2 is not supported.
  // CHECK-NEXT: */
  b = __hbeq2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbequ2 is not supported.
  // CHECK-NEXT: */
  b = __hbequ2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbge2 is not supported.
  // CHECK-NEXT: */
  b = __hbge2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbgeu2 is not supported.
  // CHECK-NEXT: */
  b = __hbgeu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbgt2 is not supported.
  // CHECK-NEXT: */
  b = __hbgt2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbgtu2 is not supported.
  // CHECK-NEXT: */
  b = __hbgtu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hble2 is not supported.
  // CHECK-NEXT: */
  b = __hble2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbleu2 is not supported.
  // CHECK-NEXT: */
  b = __hbleu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hblt2 is not supported.
  // CHECK-NEXT: */
  b = __hblt2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbltu2 is not supported.
  // CHECK-NEXT: */
  b = __hbltu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbne2 is not supported.
  // CHECK-NEXT: */
  b = __hbne2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hbneu2 is not supported.
  // CHECK-NEXT: */
  b = __hbneu2(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hequ2 is not supported.
  // CHECK-NEXT: */
  __hequ2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hgeu2 is not supported.
  // CHECK-NEXT: */
  __hgeu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hgtu2 is not supported.
  // CHECK-NEXT: */
  __hgtu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hleu2 is not supported.
  // CHECK-NEXT: */
  __hleu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hltu2 is not supported.
  // CHECK-NEXT: */
  __hltu2(h2, h2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of __hneu2 is not supported.
  // CHECK-NEXT: */
  __hneu2(h2, h2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i0f is not supported.
  // CHECK-NEXT: */
  f = cyl_bessel_i0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i1f is not supported.
  // CHECK-NEXT: */
  f = cyl_bessel_i1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfcinvf is not supported.
  // CHECK-NEXT: */
  f = erfcinvf(f);
  // CHECK: f = sycl::exp(f*f)*sycl::erfc(f);
  f = erfcxf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfinvf is not supported.
  // CHECK-NEXT: */
  f = erfinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j0f is not supported.
  // CHECK-NEXT: */
  f = j0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j1f is not supported.
  // CHECK-NEXT: */
  f = j1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of jnf is not supported.
  // CHECK-NEXT: */
  f = jnf(i, f);

  // CHECK: f = sycl::fast_length(sycl::float3(f, f, f));
  f = norm3df(f, f, f);
  // CHECK: f = sycl::fast_length(sycl::float4(f, f, f, f));
  f = norm4df(f, f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdff is not supported.
  // CHECK-NEXT: */
  f = normcdff(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdfinvf is not supported.
  // CHECK-NEXT: */
  f = normcdfinvf(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normf is not supported.
  // CHECK-NEXT: */
  f = normf(i, &f);
  // CHECK: f = sycl::native::recip((float)sycl::cbrt(f));
  f = rcbrtf(f);
  // CHECK: f = sycl::native::recip(sycl::fast_length(sycl::float3(f, f, f)));
  f = rnorm3df(f, f, f);
  // CHECK: f = sycl::native::recip(sycl::fast_length(sycl::float4(f, f, f, f)));
  f = rnorm4df(f, f, f, f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of rnormf is not supported.
  // CHECK-NEXT: */
  f = rnormf(i, &f);
  // CHECK: f = f*(2<<l);
  f = scalblnf(f, l);
  // CHECK: f = f*(2<<i);
  f = scalbnf(f, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y0f is not supported.
  // CHECK-NEXT: */
  f = y0f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y1f is not supported.
  // CHECK-NEXT: */
  f = y1f(f);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of ynf is not supported.
  // CHECK-NEXT: */
  f = ynf(i, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i0 is not supported.
  // CHECK-NEXT: */
  d = cyl_bessel_i0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cyl_bessel_i1 is not supported.
  // CHECK-NEXT: */
  d = cyl_bessel_i1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfcinv is not supported.
  // CHECK-NEXT: */
  d = erfcinv(d);
  // CHECK: d = sycl::exp(d*d)*sycl::erfc(d);
  d = erfcx(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of erfinv is not supported.
  // CHECK-NEXT: */
  d = erfinv(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j0 is not supported.
  // CHECK-NEXT: */
  d = j0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of j1 is not supported.
  // CHECK-NEXT: */
  d = j1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of jn is not supported.
  // CHECK-NEXT: */
  d = jn(i, d);
  // CHECK: d = dpct::fast_length((float *)&d, i);
  d = norm(i, &d);
  // CHECK: d = sycl::fast_length(sycl::float3(d, d, d));
  d = norm3d(d, d, d);
  // CHECK: d = sycl::fast_length(sycl::float4(d, d, d, d));
  d = norm4d(d, d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdf is not supported.
  // CHECK-NEXT: */
  d = normcdf(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of normcdfinv is not supported.
  // CHECK-NEXT: */
  d = normcdfinv(d);
  // CHECK: d = sycl::native::recip((float)sycl::cbrt(d));
  d = rcbrt(d);
  // CHECK: d = sycl::native::recip(sycl::fast_length(sycl::float3(d, d, d)));
  d = rnorm3d(d, d, d);
  // CHECK: d = sycl::native::recip(sycl::fast_length(sycl::float4(d, d, d, d)));
  d = rnorm4d(d, d, d, d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of rnorm is not supported.
  // CHECK-NEXT: */
  d = rnorm(i, &d);
  // CHECK: d = d*(2<<l);
  d = scalbln(d, l);
  // CHECK: d = d*(2<<i);
  d = scalbn(d, i);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y0 is not supported.
  // CHECK-NEXT: */
  d = y0(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of y1 is not supported.
  // CHECK-NEXT: */
  d = y1(d);
  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of yn is not supported.
  // CHECK-NEXT: */
  d = yn(i, d);

  // CHECK: f = sycl::clamp<float>(f, 0.0f, 1.0f);
  f = __saturatef(f);

  // i = __shfl_down_sync(u, h, u, i);
  // i = __shfl_sync(u, h, u, i);
  // i = __shfl_up_sync(u, h, u, i);
  // i = __shfl_xor_sync(u, h, u, i);

  // CHECK: i = dpct::cast_double_to_int(d);
  i = __double2hiint(d);
  // CHECK: i = dpct::cast_double_to_int(d, false);
  i = __double2loint(d);
  // CHECK: d = dpct::cast_ints_to_double(i, i);
  d = __hiloint2double(i, i);

  // CHECK: u = dpct::reverse_bits<unsigned int>(u);
  u = __brev(u);
  // CHECK: ull = dpct::reverse_bits<unsigned long long>(ull);
  ull = __brevll(ull);
  // CHECK: u = dpct::byte_level_permute(u, u, u);
  u = __byte_perm(u, u, u);
  // CHECK: i = dpct::ffs<int>(i);
  i = __ffs(i);
  // CHECK: i = dpct::ffs<long long int>(ll);
  i = __ffsll(ll);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The ((upsample(hi, lo) << (shift & 31)) >> 32) expression is used instead of the __funnelshift_l call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = ((sycl::upsample<unsigned>(u, u) << (u & 31)) >> 32);
  u = __funnelshift_l(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The ((upsample(hi, lo) << min(shift, 32)) >> 32) expression is used instead of the __funnelshift_lc call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = ((sycl::upsample<unsigned>(u, u) << sycl::min(u, 32)) >> 32);
  u = __funnelshift_lc(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The ((upsample(hi, lo) >> (shift & 31)) & 0xFFFFFFFF) expression is used instead of the __funnelshift_r call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = ((sycl::upsample<unsigned>(u, u) >> (u & 31)) & 0xFFFFFFFF);
  u = __funnelshift_r(u, u, u);
  // CHECK: /*
  // CHECK-NEXT: DPCT1098:{{[0-9]+}}: The ((upsample(hi, lo) >> min(shift, 32)) & 0xFFFFFFFF) expression is used instead of the __funnelshift_rc call. These two expressions do not provide the exact same functionality. Check the generated code for potential precision and/or performance issues.
  // CHECK-NEXT: */
  // CHECK-NEXT: u = ((sycl::upsample<unsigned>(u, u) >> sycl::min(u, 32)) & 0xFFFFFFFF);
  u = __funnelshift_rc(u, u, u);
  // CHECK: ll = sycl::mul_hi(ll, ll);
  ll = __mul64hi(ll, ll);
  // CHECK: i = sycl::rhadd(i, i);
  i = __rhadd(i, i);
  // CHECK: u = sycl::abs_diff(i, i)+u;
  u = __sad(i, i, u);
  // CHECK: u = sycl::hadd(u, u);
  u = __uhadd(u, u);
  // CHECK: u = sycl::mul24(u, u);
  u = __umul24(u, u);
  // CHECK: ull = sycl::mul_hi(ull, ull);
  ull = __umul64hi(ull, ull);
  // CHECK: u = sycl::mul_hi(u, u);
  u = __umulhi(u, u);
  // CHECK: u = sycl::rhadd(u, u);
  u = __urhadd(u, u);
  // CHECK: u = sycl::abs_diff(u, u)+u;
  u = __usad(u, u, u);
}

__global__ void testSimulation() {
  float f;
  double d;

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::floor(f + 0.5);
  f = nearbyintf(f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::floor(d + 0.5);
  d = nearbyint(d);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::hypot call is used instead of the rhypotf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = 1 / sycl::hypot(f, f);
  f = rhypotf(f, f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospif call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: f = sycl::sincos(f * DPCT_PI_F, sycl::make_ptr<float, sycl::access::address_space::private_space>(&f));
  sincospif(f, &f, &f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincospi call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: d = sycl::sincos(d * DPCT_PI, sycl::make_ptr<double, sycl::access::address_space::private_space>(&d));
  sincospi(d, &d, &d);
}

__global__ void testIntegerFunctions() {
  int i;
  unsigned u;
  long l;
  long long ll;
  unsigned long long ull;

  // CHECK: i = sycl::clz(i);
  // CHECK-NEXT: i = sycl::clz(ll);
  // CHECK-NEXT: i = sycl::hadd(i, i);
  // CHECK-NEXT: i = sycl::mul24(i, i);
  // CHECK-NEXT: i = sycl::mul_hi(i, i);
  // CHECK-NEXT: i = sycl::popcount(u);
  // CHECK-NEXT: i = sycl::popcount(ull);
  i = __clz(i);
  i = __clzll(ll);
  i = __hadd(i, i);
  i = __mul24(i, i);
  i = __mulhi(i, i);
  i = __popc(u);
  i = __popcll(ull);

  // CHECK: sycl::clz((int)u);
  // CHECK-NEXT: sycl::clz((long long)ull);
  // CHECK-NEXT: sycl::hadd((int)u, (int)u);
  // CHECK-NEXT: sycl::mul24((int)u, (int)u);
  // CHECK-NEXT: sycl::mul_hi((int)u, (int)u);
  __clz(u);
  __clzll(ull);
  __hadd(u, u);
  __mul24(u, u);
  __mulhi(u, u);

  // CHECK: i = sycl::abs(i);
  // CHECK-NEXT: l = sycl::abs(l);
  // CHECK-NEXT: ll = sycl::abs(ll);
  i = abs(i);
  l = labs(l);
  ll = llabs(ll);
}

void testTypecasts() {

}

__global__ void testConditionalOperator(float *deviceArrayFloat) {
  float &f0 = *deviceArrayFloat, &f1 = *(deviceArrayFloat + 1),
        &f2 = *(deviceArrayFloat + 2);
  // CHECK: f0 = sycl::fmax(f0 = (f1) > (f1 == 1 ? 0 : -f2) ? sycl::pow<float>(f1, 2.f) / f1 : -f1, f1 + f1 < f2
  // CHECK-NEXT:         ? ((f1) > (f1 == 1 ? 0 : -f2) ? sycl::pow<float>(f2, 2.f) / f1 : -f1)
  // CHECK-NEXT:         : -f1);
  // CHECK-NEXT: f0 = f1 > f2 ? sycl::pow<float>(f1, 2.f) / f1 : f1;
  // CHECK-NEXT: f0 = sycl::fmax(0 ? sycl::pow<float>(f1, 2.f) / f1 : f1, f2);
  f0 = fmaxf(
      f0 = (f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f1, 2.f), f1) : -f1,
      f1 + f1 < f2
          ? ((f1) > (f1 == 1 ? 0 : -f2) ? __fdividef(__powf(f2, 2.f), f1) : -f1)
          : -f1);
  f0 = f1 > f2 ? __fdividef(__powf(f1, 2.f), f1) : f1;
  f0 = fmaxf(0 ? __fdividef(__powf(f1, 2.f), f1) : f1, f2);
}

int main() {
  testDouble();
  testFloat();
  testTypecasts();
}

// Host max/min functions with integer parameters are in <algorithm> instead of <cmath>, so we need to
// migrate them to std versions and do necessary casts.
// The following migration is to make the resulted code compilable by dpcpp
// rule1: in pure __host__ functions, math functions are migrated to std alternatives
// rule2: math functions in __device__ or __global__ functions are always migrated to sycl alternatives
// rule3: functions in std namespace always remain untouched

// CHECK:  int foo(int i, int j) {
// CHECK-NEXT:   return std::max(i, j) + std::min(i, j);
// CHECK-NEXT: }
__host__ int foo(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo(float f, float g) {
// CHECK-NEXT:   return fmaxf(f, g) + fminf(f, g);
// CHECK-NEXT: }
__host__ float foo(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int foo2(int i, int j) {
// CHECK-NEXT:   return sycl::max(i, j) + sycl::min(i, j);
// CHECK-NEXT: }
__device__ int foo2(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float foo2(float f, float g) {
// CHECK-NEXT:   return sycl::max(f, g) + sycl::min(f, g);
// CHECK-NEXT: }
__device__ float foo2(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int  foo3(int i, int j) {
// CHECK-NEXT:   return sycl::max(i, j) + sycl::min(i, j);
// CHECK-NEXT: }
__device__ int __host__ foo3(int i, int j) {
  return max(i, j) + min(i, j);
}

// CHECK:  float  foo3(float f, float g) {
// CHECK-NEXT:   return sycl::max(f, g) + sycl::min(f, g);
// CHECK-NEXT: }
__device__ float __host__ foo3(float f, float g) {
  return max(f, g) + min(f, g);
}

// CHECK:  int bar(short i, long j) {
// CHECK-NEXT:   return std::max<long>(i, j) + std::min<long>(i, j);
// CHECK-NEXT: }
__host__ int bar(short i, long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned short i, unsigned long j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned short i, unsigned long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned short i, long j) {
// CHECK-NEXT:   return max(i, j) + min(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned short i, long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(long i, unsigned short j) {
// CHECK-NEXT:   return max(i, j) + min(i, j);
// CHECK-NEXT: }
__host__ int bar(long i, unsigned short j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(short i, unsigned long j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(short i, unsigned long j) {
  return max(i, j) + min(i, j);
}

// CHECK:  int bar(unsigned long i, short j) {
// CHECK-NEXT:   return std::max<unsigned long>(i, j) + std::min<unsigned long>(i, j);
// CHECK-NEXT: }
__host__ int bar(unsigned long i, short j) {
  return max(i, j) + min(i, j);
}

typedef int INT;
typedef unsigned UINT;
using int_t = int;
using uint_t = unsigned;

// CHECK: int foo(UINT i, INT j) {
// CHECK-NEXT:   return std::max<UINT>(i, j) + std::min<UINT>(i, j);
// CHECK-NEXT: }
int foo(UINT i, INT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int foo(INT i, UINT j) {
// CHECK-NEXT:   return std::max<UINT>(i, j) + std::min<UINT>(i, j);
// CHECK-NEXT: }
int foo(INT i, UINT j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(uint_t i, int_t j) {
// CHECK-NEXT:   return std::max<uint_t>(i, j) + std::min<uint_t>(i, j);
// CHECK-NEXT: }
int bar(uint_t i, int_t j) {
  return max(i, j) + min(i, j);
}

// CHECK: int bar(int_t i, uint_t j) {
// CHECK-NEXT:   return std::max<uint_t>(i, j) + std::min<uint_t>(i, j);
// CHECK-NEXT: }
int bar(int_t i, uint_t j) {
  return max(i, j) + min(i, j);
}

__device__ void test_pow() {
  int i;
  float f;
  double d;

  // CHECK: sycl::pown((float)i, i);
  pow(i, i);
  // CHECK: sycl::pown(f, i);
  pow(f, i);
  // CHECK: sycl::pown(d, i);
  pow(d, i);

  // CHECK: sycl::pow<double>(i, f);
  pow(i, f);
  // CHECK: sycl::pow<double>(f, f);
  pow(f, f);
  // CHECK: sycl::pow<double>(d, f);
  pow(d, f);

  // CHECK: sycl::pow<double>(i, d);
  pow(i, d);
  // CHECK: sycl::pow<double>(f, d);
  pow(f, d);
  // CHECK: sycl::pow<double>(d, d);
  pow(d, d);
}

__global__ void foobar(int i) {
  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_id(0), (unsigned int)i);
  max(i, threadIdx.x);
  max(i, threadIdx.y);
  max(i, threadIdx.z);
  max(threadIdx.x, i);
  max(threadIdx.y, i);
  max(threadIdx.z, i);

  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_group(0), (unsigned int)i);
  max(i, blockIdx.x);
  max(i, blockIdx.y);
  max(i, blockIdx.z);
  max(blockIdx.x, i);
  max(blockIdx.y, i);
  max(blockIdx.z, i);

  // CHECK: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: sycl::max((unsigned int)i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(2), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(1), (unsigned int)i);
  // CHECK-NEXT: sycl::max((unsigned int)item_ct1.get_local_range(0), (unsigned int)i);
  max(i, blockDim.x);
  max(i, blockDim.y);
  max(i, blockDim.z);
  max(blockDim.x, i);
  max(blockDim.y, i);
  max(blockDim.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_id(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_id(0), (unsigned int)i);
  min(i, threadIdx.x);
  min(i, threadIdx.y);
  min(i, threadIdx.z);
  min(threadIdx.x, i);
  min(threadIdx.y, i);
  min(threadIdx.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_group(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_group(0), (unsigned int)i);
  min(i, blockIdx.x);
  min(i, blockIdx.y);
  min(i, blockIdx.z);
  min(blockIdx.x, i);
  min(blockIdx.y, i);
  min(blockIdx.z, i);

  // CHECK: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(2));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(1));
  // CHECK-NEXT: sycl::min((unsigned int)i, (unsigned int)item_ct1.get_local_range(0));
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(2), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(1), (unsigned int)i);
  // CHECK-NEXT: sycl::min((unsigned int)item_ct1.get_local_range(0), (unsigned int)i);
  min(i, blockDim.x);
  min(i, blockDim.y);
  min(i, blockDim.z);
  min(blockDim.x, i);
  min(blockDim.y, i);
  min(blockDim.z, i);
}

void do_migration() {
  int i, j;
  // CHECK: std::max(i, j);
  max(i, j);
}
__global__ void do_migration2() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__device__ void do_migration3() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
__host__ __device__ void do_migration4() {
  int i, j;
  // CHECK: sycl::max(i, j);
  max(i, j);
}
int max(int i, int j) {
  return i > j ? i : j;
}
namespace t {
int max(int i, int j) {
  return i > j ? i : j;
}
}
void no_migration() {
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}
void no_migration2() {
  int i, j;
  // CHECK: t::max(i, j);
  t::max(i, j);
}
void no_migration3() {
  int i, j;
  // CHECK: std::max(i, j);
  std::max(i, j);
}
__host__ void no_migration4() {
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}

void ns() {
  using namespace std;
  int i, j;
  // CHECK: max(i, j);
  max(i, j);
}

void no_migration5() {
  float f;
  int i;

  //CHECK: std::max(i, i);
  //CHECK-NEXT: std::min(i, i);
  //CHECK-NEXT: std::fabs(f);
  //CHECK-NEXT: std::frexpf(f, &i);
  //CHECK-NEXT: std::modff(f, &f);
  //CHECK-NEXT: std::nearbyintf(f);
  //CHECK-NEXT: std::remquof(f, f, &i);
  //CHECK-NEXT: std::acosf(f);
  //CHECK-NEXT: std::acoshf(f);
  //CHECK-NEXT: std::asinf(f);
  //CHECK-NEXT: std::asinhf(f);
  //CHECK-NEXT: std::abs(f);
  //CHECK-NEXT: std::frexp(f, &i);
  //CHECK-NEXT: std::modf(f, &f);
  //CHECK-NEXT: std::nearbyint(f);
  //CHECK-NEXT: std::remquo(f, f, &i);
  //CHECK-NEXT: std::acos(f);
  //CHECK-NEXT: std::acosh(f);
  //CHECK-NEXT: std::asin(f);
  //CHECK-NEXT: std::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);

  int64_t a;
  //CHECK: std::max<int64_t>(a, 1);
  std::max<int64_t>(a, 1);
}

__device__ void do_migration5() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__global__ void do_migration6() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ __host__ void do_migration7() {
  float f;
  int i;

  //CHECK: sycl::max(i, i);
  //CHECK-NEXT: sycl::min(i, i);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexpf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modff call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<float, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyintf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquof call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  //CHECK-NEXT: sycl::fabs(f);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::frexp(f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::modf call is used instead of the modf call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::modf(f, sycl::make_ptr<double, sycl::access::address_space::private_space>(&f));
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::floor call is used instead of the nearbyint call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::floor(f + 0.5);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1017:{{[0-9]+}}: The sycl::remquo call is used instead of the remquo call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::remquo(f, f, sycl::make_ptr<int, sycl::access::address_space::private_space>(&i));
  //CHECK-NEXT: sycl::acos(f);
  //CHECK-NEXT: sycl::acosh(f);
  //CHECK-NEXT: sycl::asin(f);
  //CHECK-NEXT: sycl::asinh(f);
  std::max(i, i);
  std::min(i, i);
  std::fabs(f);
  std::frexpf(f, &i);
  std::modff(f, &f);
  std::nearbyintf(f);
  std::remquof(f, f, &i);
  std::acosf(f);
  std::acoshf(f);
  std::asinf(f);
  std::asinhf(f);
  std::abs(f);
  std::frexp(f, &i);
  std::modf(f, &f);
  std::nearbyint(f);
  std::remquo(f, f, &i);
  std::acos(f);
  std::acosh(f);
  std::asin(f);
  std::asinh(f);
}

__device__ void test_recursive_unary() {
  int i, j, k;
  // CHECK: sycl::max(-sycl::max(-sycl::abs(i), j), k);
  max(-max(-abs(i), j), k);
}

__device__ void do_math(int i, int j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt((double)i);
  sqrt(i);
  // CHECK: sycl::fmod((double)i, (double)j);
  fmod(i, j);
  // CHECK: sycl::sin((double)i);
  sin(i);
  // CHECK: sycl::cos((double)i);
  cos(i);
}

__device__ void do_math(float i, float j) {
  // CHECK: sycl::sqrt(i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}

__device__ void do_math(double i, double j) {
  // CHECK: sycl::sqrt((float)i);
  sqrtf(i);
  // CHECK: sycl::sqrt(i);
  sqrt(i);
  // CHECK: sycl::fmod(i, j);
  fmod(i, j);
  // CHECK: sycl::sin(i);
  sin(i);
  // CHECK: sycl::cos(i);
  cos(i);
}

__global__ void k() {
  float f;

  char c;
  unsigned char uc;
  short s;
  unsigned short us;
  int i;
  unsigned int ui;
  long l;
  unsigned long ul;
  long long ll;
  unsigned long long ull;

  // CHECK: f * f;
  pow(f, 2);
  // CHECK: sycl::pown(f, 3);
  pow(f, 3);
  // CHECK: f * f;
  powf(f, 2);
  // CHECK: sycl::pown(f, 3);
  powf(f, 3);
  // CHECK: f * f;
  __powf(f, 2);
  // CHECK: sycl::pown(f, 3);
  __powf(f, 3);

  // CHECK: sycl::pown(f, (int)c);
  pow(f, c);
  // CHECK: sycl::pown(f, (int)uc);
  pow(f, uc);
  // CHECK: sycl::pown(f, (int)s);
  pow(f, s);
  // CHECK: sycl::pown(f, (int)us);
  pow(f, us);
  // CHECK: sycl::pown(f, i);
  pow(f, i);
  // CHECK: sycl::pown(f, (int)ui);
  pow(f, ui);
  // CHECK: sycl::pow<double>(f, l);
  pow(f, l);
  // CHECK: sycl::pow<double>(f, ul);
  pow(f, ul);
  // CHECK: sycl::pow<double>(f, ll);
  pow(f, ll);
  // CHECK: sycl::pow<double>(f, ull);
  pow(f, ull);

  // CHECK: sycl::pown(f, (int)c);
  powf(f, c);
  // CHECK: sycl::pown(f, (int)uc);
  powf(f, uc);
  // CHECK: sycl::pown(f, (int)s);
  powf(f, s);
  // CHECK: sycl::pown(f, (int)us);
  powf(f, us);
  // CHECK: sycl::pown(f, i);
  powf(f, i);
  // CHECK: sycl::pown(f, (int)ui);
  powf(f, ui);
  // CHECK: sycl::pow<float>(f, l);
  powf(f, l);
  // CHECK: sycl::pow<float>(f, ul);
  powf(f, ul);
  // CHECK: sycl::pow<float>(f, ll);
  powf(f, ll);
  // CHECK: sycl::pow<float>(f, ull);
  powf(f, ull);

  // CHECK: sycl::pown(f, (int)c);
  __powf(f, c);
  // CHECK: sycl::pown(f, (int)uc);
  __powf(f, uc);
  // CHECK: sycl::pown(f, (int)s);
  __powf(f, s);
  // CHECK: sycl::pown(f, (int)us);
  __powf(f, us);
  // CHECK: sycl::pown(f, i);
  __powf(f, i);
  // CHECK: sycl::pown(f, (int)ui);
  __powf(f, ui);
  // CHECK: sycl::pow<float>(f, l);
  __powf(f, l);
  // CHECK: sycl::pow<float>(f, ul);
  __powf(f, ul);
  // CHECK: sycl::pow<float>(f, ll);
  __powf(f, ll);
  // CHECK: sycl::pow<float>(f, ull);
  __powf(f, ull);

}

__global__ void k2() {
  int i, i2;
  unsigned u, u1, u2;
  float f0, f1, f2, f3;
  double d0, d1, d2, d3;
  long l, l2;
  long long ll, ll2;
  unsigned long long ull, ull2;

  // CHECK: sycl::exp(d0*d0)*sycl::erfc(d0);
  erfcx(d0);
  // CHECK: sycl::exp(f0*f0)*sycl::erfc(f0);
  erfcxf(f0);
  // CHECK: sycl::fast_length(sycl::float3(d0, d1, d2));
  norm3d(d0, d1, d2);
  // CHECK: sycl::fast_length(sycl::float3(f0, f1, f2));
  norm3df(f0, f1, f2);
  // CHECK: sycl::fast_length(sycl::float4(d0, d1, d2, d3));
  norm4d(d0, d1, d2, d3);
  // CHECK: sycl::fast_length(sycl::float4(f0, f1, f2, f3));
  norm4df(f0, f1, f2, f3);
  // CHECK: sycl::native::recip((float)sycl::cbrt(d0));
  rcbrt(d0);
  // CHECK: sycl::native::recip((float)sycl::cbrt(f0));
  rcbrtf(f0);
  // CHECK: sycl::native::recip(sycl::fast_length(sycl::float3(d0, d1, d2)));
  rnorm3d(d0, d1, d2);
  // CHECK: sycl::native::recip(sycl::fast_length(sycl::float3(f0, f1, f2)));
  rnorm3df(f0, f1, f2);
  // CHECK: sycl::native::recip(sycl::fast_length(sycl::float4(d0, d1, d2, d3)));
  rnorm4d(d0, d1, d2, d3);
  // CHECK: sycl::native::recip(sycl::fast_length(sycl::float4(f0, f1, f2, f3)));
  rnorm4df(f0, f1, f2, f3);
  // CHECK: d0*(2<<l);
  scalbln(d0, l);
  // CHECK: f0*(2<<l);
  scalblnf(f0, l);
  // CHECK: d0*(2<<i);
  scalbn(d0, i);
  // CHECK: f0*(2<<i);
  scalbnf(f0, i);
  // CHECK: dpct::cast_double_to_int(d0);
  __double2hiint(d0);
  // CHECK: dpct::cast_double_to_int(d0, false);
  __double2loint(d0);
  // CHECK: dpct::cast_ints_to_double(i, i2);
  __hiloint2double(i, i2);

  // CHECK: sycl::abs_diff(i, i2)+u;
  __sad(i, i2, u);
  // CHECK: sycl::abs_diff(u, u1)+u2;
  __usad(u, u1, u2);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rd(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rn(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_ru(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d0);
  __drcp_rz(d0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/(d0+d0));
  __drcp_rz(d0+d0);

  // CHECK: sycl::mul_hi(ll, ll);
  __mul64hi(ll, ll);
  // CHECK: sycl::rhadd(i, i2);
  __rhadd(i, i2);
  // CHECK: sycl::hadd(u, u2);
  __uhadd(u, u2);
  // CHECK: sycl::mul24(u, u2);
  __umul24(u, u2);
  // CHECK: sycl::mul_hi(ull, ull2);
  __umul64hi(ull, ull2);
  // CHECK: sycl::mul_hi(u, u2);
  __umulhi(u, u2);
  // CHECK: sycl::rhadd(u, u2);
  __urhadd(u, u2);

  // CHECK: u = dpct::vectorized_max<sycl::char4>(u, u2);
  u = __vmaxs4(u, u2);

  // CHECK: u = dpct::vectorized_max<sycl::ushort2>(u, u2);
  u = __vmaxu2(u, u2);

  // CHECK: u = dpct::vectorized_min<sycl::ushort2>(u, u2);
  u = __vminu2(u, u2);

  // CHECK: u = dpct::vectorized_min<sycl::uchar4>(u, u2);
  u = __vminu4(u, u2);

  // CHECK: u = dpct::vectorized_isgreater<sycl::ushort2, unsigned>(u, u2);
  u = __vcmpgtu2(u, u2);

  // CHECK: u = dpct::vectorized_isgreater<sycl::uchar4, unsigned>(u, u2);
  u = __vcmpgtu4(u, u2);

  double *a_d;
  // CHECK: 0;
  norm(0, a_d);
  // CHECK: sycl::fast_length((float)a_d[0]);
  norm(1, a_d);
  // CHECK: sycl::fast_length(sycl::float2(a_d[0], a_d[1]));
  norm(2, a_d);
  // CHECK: sycl::fast_length(sycl::float3(a_d[0], a_d[1], a_d[2]));
  norm(3, a_d);
  // CHECK: sycl::fast_length(sycl::float4(a_d[0], a_d[1], a_d[2], a_d[3]));
  norm(4, a_d);
  // CHECK: dpct::fast_length((float *)a_d, 5);
  norm(5, a_d);
}

// CHECK: #define MUL(a, b) sycl::mul24((int)a, (int)b)
#define MUL(a, b) __mul24(a, b)
__global__ void test_mul24_complicated() {
  // CHECK: unsigned int      tid = sycl::mul24((int)item_ct1.get_local_range(2), (int)item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int      tid = __mul24(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int  threadN = sycl::mul24((int)item_ct1.get_local_range(2), (int)item_ct1.get_group_range(2));
  unsigned int  threadN = __mul24(blockDim.x, gridDim.x);

  // CHECK: unsigned int     tid2 = MUL(item_ct1.get_local_range(2), item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int     tid2 = MUL(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int threadN2 = MUL(item_ct1.get_local_range(2), item_ct1.get_group_range(2));
  unsigned int threadN2 = MUL(blockDim.x, gridDim.x);
}

// CHECK: #define UMUL(a, b) sycl::mul24((unsigned int)a, (unsigned int)b)
#define UMUL(a, b) __umul24(a, b)

__global__ void test_umul24_complicated() {
  // CHECK: unsigned int      tid = sycl::mul24((unsigned int)item_ct1.get_local_range(2), (unsigned int)item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int      tid = __umul24(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int  threadN = sycl::mul24((unsigned int)item_ct1.get_local_range(2), (unsigned int)item_ct1.get_group_range(2));
  unsigned int  threadN = __umul24(blockDim.x, gridDim.x);

  // CHECK: unsigned int     tid2 = UMUL(item_ct1.get_local_range(2), item_ct1.get_group(2)) + item_ct1.get_local_id(2);
  unsigned int     tid2 = UMUL(blockDim.x, blockIdx.x) + threadIdx.x;
  // CHECK: unsigned int threadN2 = UMUL(item_ct1.get_local_range(2), item_ct1.get_group_range(2));
  unsigned int threadN2 = UMUL(blockDim.x, gridDim.x);
}

struct S {
  int m;
};

__device__ int fun(int i) { return i * 2; }

__device__ S fun2(int i) { return { i * 2 }; }

#define TWO 2.0

// CHECK: #define POW_TWO(x) sycl::pow<double>(x, 2.0)
#define POW_TWO(x) pow(x, 2.0)

// CHECK: #define POW(x, y) sycl::pow<double>(x, y)
#define POW(x, y) pow(x, y)

__global__ void test_side_effects() {
  int a, b[10];
  S s;
  S *sp = new S;

  // CHECK: int c = (a - b[0]) * (a - b[0]);
  int c = pow(a - b[0], 2);
  // CHECK: int d = a * a;
  int d = pow(a, 2);
  // CHECK: int e = 2 * 2;
  int e = pow(2, 2);
  // CHECK: int f = 2.0 * 2.0;
  int f = pow(2.0, 2);
  // CHECK: int g = (a ? b[0] : b[1]) * (a ? b[0] : b[1]);
  int g = pow(a ? b[0] : b[1] , 2);
  // CHECK: int h = (a >> 2) * (a >> 2);
  int h = pow(a >> 2, 2);
  // CHECK: int i = sycl::pown((float)(fun(a)), 2);
  int i = pow(fun(a), 2);
  // CHECK: int j = b[0] * b[0];
  int j = pow(b[0], 2);
  // CHECK: int k = (a + b[0]) * (a + b[0]);
  int k = pow((a + b[0]), 2);
  // CHECK: int l = s.m * s.m;
  int l = pow(s.m, 2);
  // CHECK: int m = sp->m * sp->m;
  int m = pow(sp->m, 2);
  // CHECK: int n = (int)a * (int)a;
  int n = pow((int)a, 2);
  // CHECK: int o = static_cast<float>(a) * static_cast<float>(a);
  int o = pow(static_cast<float>(a), 2);
  // CHECK: int p = (a & b[0]) * (a & b[0]);
  int p = pow(a & b[0], 2);
  // CHECK: int q = (a && b[0]) * (a && b[0]);
  int q = pow(a && b[0], 2);
  // CHECK: int r = sycl::pown((float)(a += b[0]), 2);
  int r = pow(a += b[0], 2);
  // CHECK: int t = sycl::pown((float)(fun2(a).m), 2);
  int t = pow(fun2(a).m, 2);
  // CHECK: int u = sycl::pown((float)(a = b[0]), 2);
  int u = pow(a = b[0], 2);

  // CHECK: int u1 = 2.0 * 2.0;
  int u1 = pow(2.0, 2.0);
  // CHECK: int v = sycl::pow<double>(2.0, 1.99999999999999999);
  int v = pow(2.0, 1.99999999999999999);
  // CHECK: int w = 2.0 * 2.0;
  int w = pow(2.0, 2.0f);
  // CHECK: int w1 = sycl::pow<double>(2.0, 2.0000000001f);
  int w1 = pow(2.0, 2.0000000001f);
  // CHECK: int w2 = sycl::pow<double>(2.0, 2.0000001f);
  int w2 = pow(2.0, 2.0000001f);
  // CHECK: int w3 = sycl::pow<double>(2.0, 2.0000000000000001);
  int w3 = pow(2.0, 2.0000000000000001);
  // CHECK: int x = 2.0 * 2.0;
  int x = pow(2.0, 2l);
  // CHECK: int y = 2.0 * 2.0;
  int y = pow(2.0, 2ul);
  // CHECK: int z = 2.0 * 2.0;
  int z = pow(2.0, 2ull);

  // CHECK: sycl::pow<double>(2.0, TWO);
  pow(2.0, TWO);
  // CHECK: POW_TWO(2.0);
  POW_TWO(2.0);
  // CHECK: POW(2.0, 2.0);
  POW(2.0, 2.0);
}

#define fp float
__device__ void foo() {
  fp d_initvalu_36;
  fp ret;
  // CHECK: ret = sycl::pow<double>(d_initvalu_36, fp(1.6));
  ret = pow(d_initvalu_36, fp(1.6));
}

// CHECK: template <typename... T>
// CHECK-NEXT: static void log(T... o)
// CHECK-NEXT: {
// CHECK-NEXT:   std::cout << "log" << std::endl;
// CHECK-NEXT: }
template <typename... T>
static void log(T... o)
{
  std::cout << "log" << std::endl;
}

// CHECK: template <typename... T>
// CHECK-NEXT: void log_log(bool cond, T... o)
// CHECK-NEXT: {
// CHECK-NEXT:   if (cond)
// CHECK-NEXT:   {
// CHECK-NEXT:     log(o...);
// CHECK-NEXT:   }
// CHECK-NEXT: }
template <typename... T>
void log_log(bool cond, T... o)
{
  if (cond)
  {
    log(o...);
  }
}

// CHECK: int test_log() {
// CHECK-NEXT:   log_log(true, "MJ", "23");
// CHECK-NEXT: }
int test_log() {
  log_log(true, "MJ", "23");
}

__device__ void bar1(double *d) {
  int i;
  double d1;
  double &d2 = *d;

  // CHECK: DPCT1081:{{[0-9]+}}: The generated code assumes that "&d2" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  // CHECK: d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(&d2));
  sincos(i, &d1, &d2);
}

__device__ void bar1(double *d, bool flag) {
  int i;
  double d1;
  double d2;
  double *d2_p;
  d2_p = &d2;

  //CHECK:/*
  //CHECK-NEXT:DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincos call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::private_space>(d2_p));
  //CHECK-NEXT:if (flag) {
  //CHECK-NEXT:  d2_p = d + 2;
  //CHECK-NEXT:}
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincos call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1081:{{[0-9]+}}: The generated code assumes that "d2_p" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  //CHECK-NEXT:*/
  //CHECK-NEXT:d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::global_space>(d2_p));
  //CHECK-NEXT:d2_p = &d2;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1017:{{[0-9]+}}: The sycl::sincos call is used instead of the sincos call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:d1 = sycl::sincos((double)i, sycl::make_ptr<double, sycl::access::address_space::private_space>(d2_p));
  sincos(i, &d1, d2_p);
  if (flag) {
    d2_p = d + 2;
  }
  sincos(i, &d1, d2_p);
  d2_p = &d2;
  sincos(i, &d1, d2_p);
}

__device__ int* get_ptr();
__device__ void bar2() {
  double d0;
  int i;
  //CHECK:/*
  //CHECK-NEXT:DPCT1017:{{[0-9]+}}: The sycl::frexp call is used instead of the frexp call. These two calls do not provide exactly the same functionality. Check the potential precision and/or performance issues for the generated code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1081:{{[0-9]+}}: The generated code assumes that "get_ptr() + i" points to the global memory address space. If it points to a local or private memory address space, replace "address_space::global" with "address_space::local" or "address_space::private".
  //CHECK-NEXT:*/
  //CHECK-NEXT:double d2 = sycl::frexp(d0, sycl::make_ptr<int, sycl::access::address_space::global_space>(get_ptr() + i));
  double d2 = frexp(d0, get_ptr() + i);
}

__device__ void foo_lambda1()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = sycl::min(x, 10) + sycl::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

__device__ __host__ void foo_lambda2()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = sycl::min(x, 10) + sycl::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

__global__ void foo_lambda3()
{
  []()
  {
    int x = 16, y = 32;
    // CHECK: int s = sycl::min(x, 10) + sycl::max(y, 64);
    int s = std::min(x, 10) + std::max(y, 64);
  }();
}

void foo_lambda4()
{
  []()
  {
    int num = 256;
    // CHECK: auto x = std::min<long long>(num, 10);
    auto x = std::min<long long>(num, 10);
    // CHECK: auto y = std::max<float>(100.0f, num);
    auto y = std::max<float>(100.0f, num);
  }();
}

void foo_lambda5()
{
  auto foo = []()
  {
    int num = 256;
    // CHECK: auto x = std::min<long long>(num, 10);
    auto x = std::min<long long>(num, 10);
    // CHECK: auto y = std::max<float>(100.0f, num);
    auto y = std::max<float>(100.0f, num);
  };
  foo();
}

void foo_lambda6()
{
  []()
  {
    []()
    {
      int num = 256;
      // CHECK: auto x = std::min<long long>(num, 10);
      auto x = std::min<long long>(num, 10);
      // CHECK: auto y = std::max<float>(100.0f, num);
      auto y = std::max<float>(100.0f, num);
    }();
  }();
}

auto static_foo = []()
{
  int num = 256;
  // CHECK: auto x = sycl::min(num, 10);
  auto x = std::min(num, 10);
  // CHECK: auto y = sycl::max(100, num);
  auto y = std::max(100, num);
};
void foo_lambda7()
{
  static_foo();
}

__device__ void qualified(double d1,
                          const double d2,
			  volatile double d3,
			  const volatile double d4) {
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rd(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rd(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rd(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rd(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rn(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rn(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rn(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rn(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_ru(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_ru(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_ru(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_ru(d4);

  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d1);
  __drcp_rz(d1);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d2);
  __drcp_rz(d2);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d3);
  __drcp_rz(d3);
  // CHECK: /*
  // CHECK-NEXT: DPCT1013:{{[0-9]+}}: The rounding mode could not be specified and the generated code may have different precision then the original code. Verify the correctness. SYCL math built-ins rounding mode is aligned with OpenCL C 1.2 standard.
  // CHECK-NEXT: */
  // CHECK-NEXT: (1.0/d4);
  __drcp_rz(d4);
}
