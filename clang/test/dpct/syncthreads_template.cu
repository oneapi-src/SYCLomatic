
#include <hip/hip_runtime.h>
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none -out-root %T/syncthreads_template %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/syncthreads_template/syncthreads_template.dp.cpp

// Unsupport template kernel function
template<class T>
struct S1 {
  T data;
};
template<class Q>
__global__ void test1(S1<Q> s1) {
  s1.data;
  // CHECK:item_ct1.barrier();
  __syncthreads();
}

template<class Q>
__global__ void test2() {
  // CHECK:item_ct1.barrier();
  __syncthreads();
}
