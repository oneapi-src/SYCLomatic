// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.2, v10.0
// RUN: dpct --format-range=none --usm-level=none -out-root %T/cusolverDnLn_cuda10-1 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn_cuda10-1/cusolverDnLn_cuda10-1.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<float>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<double>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda));
    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    //CHECK: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}

