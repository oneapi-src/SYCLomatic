// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/driver_context %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/driver_context/driver_context.dp.cpp
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)

int main(){

  hipDevice_t device;

  // CHECK: int ctx;
  hipCtx_t ctx;

  // CHECK: int ctx2;
  hipCtx_t ctx2;

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipInit was removed because this call is redundant in SYCL.
  // CHECK-NEXT: */
  hipInit(0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipInit was replaced with 0 because this call is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipInit(0));

  // CHECK: ctx = dpct::select_device(device);
  hipCtxCreate(&ctx, hipDeviceLmemResizeToMax, device);

  // CHECK: ctx = dpct::select_device(device);
  hipDevicePrimaryCtxRetain(&ctx, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDevicePrimaryCtxRelease was removed because this call is redundant in SYCL.
  // CHECK-NEXT: */
  hipDevicePrimaryCtxRelease(device);

  // CHECK: MY_SAFE_CALL((ctx = dpct::select_device(device), 0));
  MY_SAFE_CALL(hipCtxCreate(&ctx, hipDeviceLmemResizeToMax, device));

  // CHECK: dpct::select_device(ctx);
  hipCtxSetCurrent(ctx);

  // CHECK: MY_SAFE_CALL((dpct::select_device(ctx), 0));
  MY_SAFE_CALL(hipCtxSetCurrent(ctx));

  // CHECK: ctx2 = dpct::dev_mgr::instance().current_device_id();
  hipCtxGetCurrent(&ctx2);

  // CHECK: MY_SAFE_CALL((ctx2 = dpct::dev_mgr::instance().current_device_id(), 0));
  MY_SAFE_CALL(hipCtxGetCurrent(&ctx2));

  // CHECK: dpct::get_current_device().queues_wait_and_throw();
  hipCtxSynchronize();

  // CHECK: MY_SAFE_CALL((dpct::get_current_device().queues_wait_and_throw(), 0));
  MY_SAFE_CALL(hipCtxSynchronize());

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipCtxDestroy was removed because this call is redundant in SYCL.
  // CHECK-NEXT: */
  hipCtxDestroy(ctx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipCtxDestroy was replaced with 0 because this call is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: MY_SAFE_CALL(0);
  MY_SAFE_CALL(hipCtxDestroy(ctx2));

  return 0;
}
