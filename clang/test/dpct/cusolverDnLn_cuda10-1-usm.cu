// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.2, v10.0
// RUN: dpct --format-range=none -out-root %T/cusolverDnLn_cuda10-1-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnLn_cuda10-1-usm/cusolverDnLn_cuda10-1-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    int m = 0;
    int n = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);
    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<float>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<double>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::potri_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda));
    status = hipsolverDnSpotri_bufferSize(*cusolverH, uplo, n, &A_f, lda, &Lwork);
    status = hipsolverDnDpotri_bufferSize(*cusolverH, uplo, n, &A_d, lda, &Lwork);
    status = hipsolverDnCpotri_bufferSize(*cusolverH, uplo, n, &A_c, lda, &Lwork);
    status = hipsolverDnZpotri_bufferSize(*cusolverH, uplo, n, &A_z, lda, &Lwork);

    //CHECK: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, (float*)&A_f, lda, (float*)&workspace_f, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, (double*)&A_d, lda, (double*)&workspace_d, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, (std::complex<float>*)&A_c, lda, (std::complex<float>*)&workspace_c, Lwork);
    //CHECK-NEXT: oneapi::mkl::lapack::potri(**cusolverH, uplo, n, (std::complex<double>*)&A_z, lda, (std::complex<double>*)&workspace_z, Lwork);
    hipsolverDnSpotri(*cusolverH, uplo, n, &A_f, lda, &workspace_f, Lwork, &devInfo);
    hipsolverDnDpotri(*cusolverH, uplo, n, &A_d, lda, &workspace_d, Lwork, &devInfo);
    hipsolverDnCpotri(*cusolverH, uplo, n, &A_c, lda, &workspace_c, Lwork, &devInfo);
    hipsolverDnZpotri(*cusolverH, uplo, n, &A_z, lda, &workspace_z, Lwork, &devInfo);
}

