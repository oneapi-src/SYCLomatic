// RUN: dpct --format-range=none -out-root %T/cufft-workspace %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/cufft-workspace/cufft-workspace.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cufft-workspace/cufft-workspace.dp.cpp -o %T/cufft-workspace/cufft-workspace.dp.o %}
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int main() {
  hipfftHandle plan;
  int rank;
  long long int *n_ll;
  long long int *inembed_ll;
  long long int istride_ll;
  long long int idist_ll;
  long long int *onembed_ll;
  long long int ostride_ll;
  long long int odist_ll;
  hipfftType type;
  long long int batch_ll;
  size_t *workSize;

  int *n;
  int *inembed;
  int istride;
  int idist;
  int *onembed;
  int ostride;
  int odist;
  int batch;

  
  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], type, batch, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], n[1], type, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], n[1], n[2], type, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(rank, n, inembed, istride, idist, onembed, ostride, odist, type, batch, workSize);
  hipfftEstimate1d(n[0], type, batch, workSize);
  hipfftEstimate2d(n[0], n[1], type, workSize);
  hipfftEstimate3d(n[0], n[1], n[2], type, workSize);
  hipfftEstimateMany(rank, n, inembed, istride, idist, onembed, ostride, odist, type, batch, workSize);

  hipfftCreate(&plan);
  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], type, batch, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], n[1], type, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(n[0], n[1], n[2], type, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(rank, n, inembed, istride, idist, onembed, ostride, odist, type, batch, workSize);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "dpct::fft::fft_engine::estimate_size" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::fft::fft_engine::estimate_size(rank, n_ll, inembed_ll, istride_ll, idist_ll, onembed_ll, ostride_ll, odist_ll, type, batch_ll, workSize);
  hipfftGetSize1d(plan, n[0], type, batch, workSize);
  hipfftGetSize2d(plan, n[0], n[1], type, workSize);
  hipfftGetSize3d(plan, n[0], n[1], n[2], type, workSize);
  hipfftGetSizeMany(plan, rank, n, inembed, istride, idist, onembed, ostride, odist, type, batch, workSize);
  hipfftGetSizeMany64(plan, rank, n_ll, inembed_ll, istride_ll, idist_ll, onembed_ll, ostride_ll, odist_ll, type, batch_ll, workSize);

  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: plan->use_internal_workspace(0);
  hipfftSetAutoAllocation(plan, 0);
  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is correct.
  // CHECK-NEXT: */
  // CHECK-NEXT: plan->commit(&dpct::get_in_order_queue(), n[0], type, batch, workSize);
  hipfftMakePlan1d(plan, n[0], type, batch, workSize);
  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: plan->get_workspace_size(workSize);
  hipfftGetSize(plan, workSize);

  void *workArea;
  // CHECK: /*
  // CHECK-NEXT: DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  // CHECK-NEXT: */
  // CHECK-NEXT: plan->set_workspace(workArea);
  hipfftSetWorkArea(plan, workArea);

  hipfftDestroy(plan);
  return 0;
}
