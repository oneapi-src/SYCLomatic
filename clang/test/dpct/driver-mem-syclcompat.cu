// RUN: dpct --format-range=none -use-syclcompat -out-root %T/driver-mem-syclcompat %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --match-full-lines --input-file %T/driver-mem-syclcompat/driver-mem-syclcompat.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST %T/driver-mem-syclcompat/driver-mem-syclcompat.dp.cpp -o %T/driver-mem/driver-mem-syclcompat.dp.o %}

#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>
#include <iostream>

#define CALL(x) x

void cuCheckError(hipError_t err) {
}

int main(){
    size_t result1, result2;
    int size = 32;
    size_t count = 32;
    float* f_A;
    hipError_t r;
    // CHECK: f_A = (float *)sycl::malloc_host(size, q_ct1);
    hipHostAlloc((void **)&f_A, size, hipHostMallocMapped);


    hipDeviceptr_t p1;
    [&p1]() {
        //CHECK:p1 = 0;
        p1 = 0;
    }();

    hipDeviceptr_t p2;
    [&]() {
        //CHECK:p2 = 0;
        p2 = 0;
    }();

    // CHECK: syclcompat::device_ptr f_D = 0;
    hipDeviceptr_t f_D = 0;
    // CHECK: syclcompat::device_ptr f_D2 = 0;
    hipDeviceptr_t f_D2 = 0;
    // CHECK: int c1, c2;
    hipCtx_t c1, c2;
    // CHECK: f_D = (syclcompat::device_ptr)sycl::malloc_device(size, q_ct1);
    hipMalloc(&f_D, size);

    // CHECK: syclcompat::queue_ptr stream;
    hipStream_t stream;
    // CHECK: stream->memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, stream);
    // CHECK: q_ct1.memcpy(f_D, f_A, size);
    hipMemcpyHtoDAsync(f_D, f_A, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_A, size).wait());
    CALL(hipMemcpyHtoD(f_D, f_A, size));

    // CHECK: stream->memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, stream);
    // CHECK: q_ct1.memcpy(f_A, f_D, size);
    hipMemcpyDtoHAsync(f_A, f_D, size, 0);
    // CHECK: q_ct1.memcpy(f_A, f_D, size).wait();
    hipMemcpyDtoH(f_A, f_D, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, stream);
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    hipMemcpyDtoDAsync(f_D, f_D2, size, 0);
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = hipMemcpyDtoDAsync(f_D, f_D2, size, 0);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    hipMemcpyDtoD(f_D, f_D2, size);
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = hipMemcpyDtoD(f_D, f_D2, size);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size).wait();
    cuMemcpy(f_D, f_D2, size);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size).wait());
    CALL(cuMemcpy(f_D, f_D2, size));
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size).wait());
    r = cuMemcpy(f_D, f_D2, size);

    // CHECK: stream->memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, stream);
    // CHECK: CALL(stream->memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, stream));
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(stream->memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, stream);

    // CHECK: q_ct1.memcpy(f_D, f_D2, size);
    cuMemcpyAsync(f_D, f_D2, size, 0);
    // CHECK: CALL(q_ct1.memcpy(f_D, f_D2, size));
    CALL(cuMemcpyAsync(f_D, f_D2, size, 0));
    // CHECK: r = SYCLCOMPAT_CHECK_ERROR(q_ct1.memcpy(f_D, f_D2, size));
    r = cuMemcpyAsync(f_D, f_D2, size, 0);

#ifndef BUILD_TEST
    // CHECK: DPCT1131:{{[0-9]+}}: The migration of "cuMemcpyPeer" is not supported with SYCLcompat currently, please adjust the code manually.
    cuMemcpyPeer(f_D, c1, f_D2, c2, size);
    // CHECK: DPCT1131:{{[0-9]+}}: The migration of "cuMemcpyPeerAsync" is not supported with SYCLcompat currently, please adjust the code manually.
    cuMemcpyPeerAsync(f_D, c1, f_D2, c2, size, stream);
#endif

    unsigned int v32 = 50000;
    unsigned short v16 = 20000;
    unsigned char v8 = (unsigned char) 200;
    //CHECK: syclcompat::memset_d32(f_D, v32, size);
    //CHECK-NEXT: syclcompat::memset_d16(f_D, v16, size * 2);
    //CHECK-NEXT: syclcompat::memset(f_D, v8, size * 4);
    //CHECK-NEXT: syclcompat::memset_d32_async(f_D, v32, size, *stream);
    //CHECK-NEXT: syclcompat::memset_d16_async(f_D, v16, size * 2, *stream);
    //CHECK-NEXT: syclcompat::memset_async(f_D, v8, size * 4, *stream);
    //CHECK-NEXT: syclcompat::memset_d32(f_D, 1, v32, 4, 6);
    //CHECK-NEXT: syclcompat::memset_d16(f_D, 1, v16, 4 * 2, 6);
    //CHECK-NEXT: syclcompat::memset(f_D, 1, v8, 4 * 4, 6);
    //CHECK-NEXT: syclcompat::memset_d32_async(f_D, 1, v32, 4, 6, *stream);
    //CHECK-NEXT: syclcompat::memset_d16_async(f_D, 1, v16, 4 * 2, 6, *stream);
    //CHECK-NEXT: syclcompat::memset_async(f_D, 1, v8, 4 * 4, 6, *stream);
    hipMemsetD32(f_D, v32, size);
    hipMemsetD16(f_D, v16, size * 2);
    hipMemsetD8(f_D, v8, size * 4);
    hipMemsetD32Async(f_D, v32, size, stream);
    hipMemsetD16Async(f_D, v16, size * 2, stream);
    hipMemsetD8Async(f_D, v8, size * 4, stream);
    hipMemsetD2D32(f_D, 1, v32, 4, 6);
    hipMemsetD2D16(f_D, 1, v16, 4 * 2, 6);
    hipMemsetD2D8(f_D, 1, v8, 4 * 4, 6);
    hipMemsetD2D32Async(f_D, 1, v32, 4, 6, stream);
    hipMemsetD2D16Async(f_D, 1, v16, 4 * 2, 6, stream);
    hipMemsetD2D8Async(f_D, 1, v8, 4 * 4, 6, stream);

    // CHECK: syclcompat::experimental::memcpy_parameter cpy;
    // CHECK-NEXT: cpy.to.pitched.set_data_ptr(f_A);
    // CHECK-NEXT: cpy.to.pitched.set_pitch(20);
    // CHECK-NEXT: cpy.to.pos[1] = 10;
    // CHECK-NEXT: cpy.to.pos[0] = 15;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy.from.pitched.set_data_ptr(f_D);
    // CHECK-NEXT: cpy.from.pitched.set_pitch(20);
    // CHECK-NEXT: cpy.from.pos[1] = 10;
    // CHECK-NEXT: cpy.from.pos[0] = 15;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy.size[0] = 4;
    // CHECK-NEXT: cpy.size[1] = 7;
    hip_Memcpy2D cpy;
    cpy.dstMemoryType = hipMemoryTypeHost;
    cpy.dstHost = f_A;
    cpy.dstPitch = 20;
    cpy.dstY = 10;
    cpy.dstXInBytes = 15;

    cpy.srcMemoryType = hipMemoryTypeDevice;
    cpy.srcDevice = f_D;
    cpy.srcPitch = 20;
    cpy.srcY = 10;
    cpy.srcXInBytes = 15;

    cpy.WidthInBytes = 4;
    cpy.Height = 7;

    // CHECK: syclcompat::experimental::memcpy(cpy);
    hipMemcpyParam2D(&cpy);
    // CHECK: syclcompat::experimental::memcpy_async(cpy, *stream);
    hipMemcpyParam2DAsync(&cpy, stream);

    hipDeviceptr_t devicePtr;

    hipError_t cu_err;

    hipDeviceptr_t cuDevPtr;

    hipDevice_t cudevice;
    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: int advise = 0;
    hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

    // CHECK: syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, advise);
    hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, advise)));
    cuCheckError(hipMemAdvise(devicePtr, count, advise, cudevice));

    // CHECK: cu_err = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, advise));
    cu_err = hipMemAdvise(devicePtr, count, advise, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, (hipMemoryAdvise)1, cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, hipMemoryAdvise(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0)));
    cuCheckError(hipMemAdvise(devicePtr, count, static_cast<hipMemoryAdvise>(1), cudevice));

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: cu_err = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0));
    cu_err = hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: syclcompat::get_device(cudevice).default_queue()->mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, cudevice);

    // CHECK: /*
    // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
    // CHECK-NEXT: */
    // CHECK-NEXT: syclcompat::cpu_device().default_queue()->mem_advise(devicePtr, count, 0);
    hipMemAdvise(devicePtr, count, hipMemAdviseUnsetPreferredLocation, hipCpuDeviceId);


    hipDeviceptr_t devPtr;
    hipError_t curesult;
    // CHECK: stream->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, stream);
    // CHECK: (*&stream)->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, *&stream);
    // CHECK: curesult = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, NULL);
    // CHECK: syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100);
    hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: curesult = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault);
    // CHECK: curesult = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy);
    // CHECK: curesult = SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100));
    curesult = hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread);
    // CHECK: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamDefault));
    // CHECK: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamLegacy));
    // CHECK: cuCheckError(SYCLCOMPAT_CHECK_ERROR(syclcompat::get_device(cudevice).default_queue()->prefetch(devPtr, 100)));
    cuCheckError(hipMemPrefetchAsync (devPtr, 100, cudevice, hipStreamPerThread));

    // CHECK: syclcompat::experimental::memcpy_parameter cpy2;
    // CHECK: /*
    // CHECK-NEXT: DPCT1131:{{[0-9]+}}: The migration of "hipArray_t" is not supported with SYCLcompat currently, please adjust the code manually.
    // CHECK-NEXT: */
    // CHECK-NEXT: hipArray_t ca;
    // CHECK-NEXT: cpy2.to.image = ca;
    // CHECK-NEXT: cpy2.to.pitched.set_pitch(5);
    // CHECK-NEXT: cpy2.to.pitched.set_y(4);
    // CHECK-NEXT: cpy2.to.pos[1] = 3;
    // CHECK-NEXT: cpy2.to.pos[2] = 2;
    // CHECK-NEXT: cpy2.to.pos[0] = 1;
    // CHECK: cpy2.from.pitched.set_data_ptr(f_A);
    // CHECK-NEXT: cpy2.from.pitched.set_pitch(5);
    // CHECK-NEXT: cpy2.from.pitched.set_y(4);
    // CHECK-NEXT: cpy2.from.pos[1] = 3;
    // CHECK-NEXT: cpy2.from.pos[2] = 2;
    // CHECK-NEXT: cpy2.from.pos[0] = 1;
    // CHECK-EMPTY:
    // CHECK-NEXT: cpy2.size[0] = 3;
    // CHECK-NEXT: cpy2.size[1] = 2;
    // CHECK-NEXT: cpy2.size[2] = 1;
    HIP_MEMCPY3D cpy2;
#ifndef BUILD_TEST
    hipArray_t ca;
    cpy2.dstMemoryType = hipMemoryTypeArray;
    cpy2.dstArray = ca;
    cpy2.dstPitch = 5;
    cpy2.dstHeight = 4;
    cpy2.dstY = 3;
    cpy2.dstZ = 2;
    cpy2.dstXInBytes = 1;
    cpy2.dstLOD = 0;
#endif

    cpy2.srcMemoryType = hipMemoryTypeHost;
    cpy2.srcHost = f_A;
    cpy2.srcPitch = 5;
    cpy2.srcHeight = 4;
    cpy2.srcY = 3;
    cpy2.srcZ = 2;
    cpy2.srcXInBytes = 1;
    cpy2.srcLOD = 0;

    cpy2.WidthInBytes = 3;
    cpy2.Height = 2;
    cpy2.Depth = 1;

    // CHECK: syclcompat::experimental::memcpy(cpy2);
    hipDrvMemcpy3D(&cpy2);

    hipStream_t cs;
    // CHECK: syclcompat::experimental::memcpy_async(cpy2, *cs);
    hipDrvMemcpy3DAsync(&cpy2, cs);

    float *h_A = (float *)malloc(100);
    // CHECK:sycl::free(h_A, q_ct1);
    hipHostFree(h_A);
    // CHECK:sycl::free(f_D, q_ct1);
    hipFree(f_D);

    unsigned int flags;
    int host;


    // CHECK: flags = 0;
    hipHostGetFlags(&flags, &host);
    // CHECK: cuCheckError(SYCLCOMPAT_CHECK_ERROR(flags = 0));
    cuCheckError(hipHostGetFlags(&flags, &host));

    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostRegister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostRegister(h_A, count, flags);
    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostRegister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT: cuCheckError(0);
    cuCheckError(hipHostRegister(h_A, count, flags));


    // CHECK:  /*
    // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipHostUnregister was removed because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    hipHostUnregister(h_A);

    // CHECK:  /*
    // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipHostUnregister was replaced with 0 because SYCL currently does not support registering of existing host memory for use by device. Use USM to allocate memory for use by host and device.
    // CHECK-NEXT: */
    // CHECK-NEXT:cuCheckError(0);
    cuCheckError(hipHostUnregister(h_A));
    return 0;
}
