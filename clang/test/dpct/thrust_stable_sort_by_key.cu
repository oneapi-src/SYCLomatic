// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct -out-root %T/thrust_stable_sort_by_key %s --cuda-include-path="%cuda-path/include" --usm-level=none
// RUN: FileCheck --input-file %T/thrust_stable_sort_by_key/thrust_stable_sort_by_key.dp.cpp --match-full-lines %s

#include <thrust/sort.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>

int main(void) {

  thrust::device_vector<int> AD(4);
  thrust::device_vector<int> BD(4);
  thrust::host_vector<int> AH(4);
  thrust::host_vector<int> BH(4);
  
  int *h_ptr;
  int *d_ptr;

  h_ptr = (int*)std::malloc(20 * sizeof(int));
  hipMalloc(&d_ptr, 20 * sizeof(int));

/*******************************************************************************************
 1. Test stable_sort_by_key
 2. Test four VERSIONs (with/without exec argument) AND (with/without comparator argument)
 3. Test each VERSION with (device_vector/host_vector/malloc-ed memory/hipMalloc-ed memory)
 *******************************************************************************************/

/*********** stable_sort_by_key ***********************************************************************************************************************************************/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::stable_sort(oneapi::dpl::execution::seq, AH.begin(), AH.end(), BH.begin());
// CHECK-NEXT:dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin());
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr + 4)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(BH.begin()));
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, BH.begin());
// CHECK-NEXT:};
// CHECK-NEXT:if (dpct::is_device_ptr(d_ptr + 4)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(d_ptr), dpct::device_pointer<int>(d_ptr + 4), dpct::device_pointer<>(BD.begin()));
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, d_ptr, d_ptr + 4, BD.begin());
// CHECK-NEXT:};
  // VERSION                                 first       last      result
  thrust::stable_sort_by_key(                AH.begin(), AH.end(), BH.begin());
  thrust::stable_sort_by_key(                AD.begin(), AD.end(), BD.begin());
  thrust::stable_sort_by_key(                h_ptr,      h_ptr+4,  BH.begin());
  thrust::stable_sort_by_key(                d_ptr,      d_ptr+4,  BD.begin());

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::stable_sort(oneapi::dpl::execution::seq, AH.begin(), AH.end(), BH.begin(), std::greater<int>());
// CHECK-NEXT:dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), std::greater<int>());
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr + 4)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(BH.begin()), std::greater<int>());
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, BH.begin(), std::greater<int>());
// CHECK-NEXT:};
// CHECK-NEXT:if (dpct::is_device_ptr(d_ptr + 4)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(d_ptr), dpct::device_pointer<int>(d_ptr + 4), dpct::device_pointer<>(BD.begin()), std::greater<int>());
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, d_ptr, d_ptr + 4, BD.begin(), std::greater<int>());
// CHECK-NEXT:};
  // VERSION                                 first       last      result      comparator
  thrust::stable_sort_by_key(                AH.begin(), AH.end(), BH.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(                AD.begin(), AD.end(), BD.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(                h_ptr,      h_ptr+4,  BH.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(                d_ptr,      d_ptr+4,  BD.begin(), thrust::greater<int>());


//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::stable_sort(oneapi::dpl::execution::seq, AH.begin(), AH.end(), BH.begin());
// CHECK-NEXT:dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin());
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(BH.begin()));
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, BH.begin());
// CHECK-NEXT:};
// CHECK-NEXT:if (dpct::is_device_ptr(d_ptr)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(d_ptr), dpct::device_pointer<int>(d_ptr + 4), dpct::device_pointer<>(BD.begin()));
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, d_ptr, d_ptr + 4, BD.begin());
// CHECK-NEXT:};
  // VERSION                 exec            first       last      result
  thrust::stable_sort_by_key(thrust::host,   AH.begin(), AH.end(), BH.begin());
  thrust::stable_sort_by_key(thrust::device, AD.begin(), AD.end(), BD.begin());
  thrust::stable_sort_by_key(thrust::host,   h_ptr,      h_ptr+4,  BH.begin());
  thrust::stable_sort_by_key(thrust::device, d_ptr,      d_ptr+4,  BD.begin());

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::stable_sort(oneapi::dpl::execution::seq, AH.begin(), AH.end(), BH.begin(), std::greater<int>());
// CHECK-NEXT:dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), BD.begin(), std::greater<int>());
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(BH.begin()), std::greater<int>());
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, BH.begin(), std::greater<int>());
// CHECK-NEXT:};
// CHECK-NEXT:if (dpct::is_device_ptr(d_ptr)) {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(d_ptr), dpct::device_pointer<int>(d_ptr + 4), dpct::device_pointer<>(BD.begin()), std::greater<int>());
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::stable_sort(oneapi::dpl::execution::seq, d_ptr, d_ptr + 4, BD.begin(), std::greater<int>());
// CHECK-NEXT:};
  // VERSION                 exec            first       last      result      comparator
  thrust::stable_sort_by_key(thrust::host,   AH.begin(), AH.end(), BH.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(thrust::device, AD.begin(), AD.end(), BD.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(thrust::host,   h_ptr,      h_ptr+4,  BH.begin(), thrust::greater<int>());
  thrust::stable_sort_by_key(thrust::device, d_ptr,      d_ptr+4,  BD.begin(), thrust::greater<int>());

  return 0;
}
