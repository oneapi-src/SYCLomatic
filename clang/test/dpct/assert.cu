// RUN: dpct --format-range=none -out-root %T/assert %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/assert/assert.dp.cpp
#include "hip/hip_runtime.h"


#include <stdio.h>

__global__ void kernel_assert(int *d_ptr, int length) {
  // CHECK: assert(0);
  // CHECK-NEXT: assert(0);
  __assert_fail("", "", 1, "");  
  __assertfail("", "", 1, "", sizeof(char));
}

__device__ void device_assert(int *d_ptr, int length) {
  // CHECK: assert(0);
  // CHECK-NEXT: assert(0);
  __assert_fail("", "", 1, "");  
  __assertfail("", "", 1, "", sizeof(char));
}

