#include <hip/hip_cooperative_groups.h>
#include <cooperative_groups/reduce.h>
namespace cg = cooperative_groups;


__device__ void reduceBlock(double *sdata, const cg::thread_block &cta) {
  const unsigned int tid = cta.thread_rank();
  cg::thread_block_tile<32> tile32 = cg::tiled_partition<32>(cta);

  cg::reduce(tile32, sdata[tid], cg::plus<double>());  
  cg::sync(cta);

}
