// RUN: dpct --format-range=none -out-root %T/texture_global_array %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture_global_array/texture_global_array.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/texture_global_array/texture_global_array.dp.cpp -o %T/texture_global_array/texture_global_array.dp.o %}

#include "hip/hip_runtime.h"


#define MAX_INSTANCES 2

// CHECK: dpct::image_data resDescInput[MAX_INSTANCES];
// CHECK-NEXT: dpct::sampling_info texDescInput[MAX_INSTANCES];
// CHECK-NEXT: dpct::image_matrix *d_Input[MAX_INSTANCES] = {NULL};
// CHECK-NEXT: dpct::image_wrapper_base_p tex_Input[MAX_INSTANCES] = {NULL};
hipResourceDesc resDescInput[MAX_INSTANCES];
hipTextureDesc texDescInput[MAX_INSTANCES];
hipArray *d_Input[MAX_INSTANCES] = {NULL};
hipTextureObject_t tex_Input[MAX_INSTANCES] = {NULL};

void createTestTexture(int instance, unsigned char *d_In, int rSize, int pSize,
                       int pPitch) {
  // CHECK: memset(&resDescInput[instance], 0, sizeof(resDescInput[instance]));
  // CHECK-NEXT: resDescInput[instance].set_data_type(dpct::image_data_type::pitch);
  // CHECK-NEXT: resDescInput[instance].set_data_ptr(d_In);
  // CHECK-NEXT: dpct::image_channel channelDesc =
  // CHECK-NEXT:     /*
  // CHECK-NEXT:     DPCT1059:{{[0-9]+}}: SYCL only supports 4-channel image format. Adjust the code.
  // CHECK-NEXT:     */
  // CHECK-NEXT:     dpct::image_channel(8, 0, 0, 0, dpct::image_channel_data_type::unsigned_int);
  // CHECK-NEXT: resDescInput[instance].set_channel(channelDesc);
  // CHECK-NEXT: resDescInput[instance].set_y(pSize);
  // CHECK-NEXT: resDescInput[instance].set_x(rSize);
  // CHECK-NEXT: resDescInput[instance].set_pitch(pPitch);
  memset(&resDescInput[instance], 0, sizeof(resDescInput[instance]));
  resDescInput[instance].resType = hipResourceTypePitch2D;
  resDescInput[instance].res.pitch2D.devPtr = d_In;
  hipChannelFormatDesc channelDesc =
      hipCreateChannelDesc(8, 0, 0, 0, hipChannelFormatKindUnsigned);
  resDescInput[instance].res.pitch2D.desc = channelDesc;
  resDescInput[instance].res.pitch2D.height = pSize;
  resDescInput[instance].res.pitch2D.width = rSize;
  resDescInput[instance].res.pitch2D.pitchInBytes = pPitch;

  // CHECK: memset(&texDescInput[instance], 0, sizeof(texDescInput[instance]));
  // CHECK-NEXT: texDescInput[instance].set(sycl::filtering_mode::linear);
  // CHECK-NEXT: texDescInput[instance].set(sycl::coordinate_normalization_mode::unnormalized);
  // CHECK-NEXT: texDescInput[instance].set(sycl::addressing_mode::clamp);
  // CHECK-NEXT: texDescInput[instance].set(sycl::addressing_mode::clamp);
  // CHECK-NEXT: if (tex_Input[instance] != NULL) {
  // CHECK-NEXT:   delete tex_Input[instance];
  // CHECK-NEXT:   tex_Input[instance] = NULL;
  // CHECK-NEXT: }
  // CHECK-NEXT: tex_Input[instance] = dpct::create_image_wrapper(resDescInput[instance], texDescInput[instance]);
  memset(&texDescInput[instance], 0, sizeof(texDescInput[instance]));
  texDescInput[instance].filterMode = hipFilterModeLinear;
  texDescInput[instance].normalizedCoords = false;
  texDescInput[instance].addressMode[0] = hipAddressModeBorder;
  texDescInput[instance].addressMode[1] = hipAddressModeBorder;
  if (tex_Input[instance] != NULL) {
    hipDestroyTextureObject(tex_Input[instance]);
    tex_Input[instance] = NULL;
  }
  hipCreateTextureObject(&tex_Input[instance], &resDescInput[instance],
                          &texDescInput[instance], NULL);
}

void createTestTextureAlternative(int instance, unsigned char *d_In, int rSize,
                                  int pSize, int pPitch) {
  // CHECK: memset(&resDescInput[instance], 0, sizeof(resDescInput[instance]));
  // CHECK-NEXT: resDescInput[instance].set_data_type(dpct::image_data_type::matrix);
  // CHECK-NEXT: resDescInput[instance].set_data_ptr(d_Input[instance]);
  // CHECK-NEXT: memset(&texDescInput[instance], 0, sizeof(texDescInput[instance]));
  // CHECK-NEXT: texDescInput[instance].set(sycl::filtering_mode::linear);
  // CHECK-NEXT: texDescInput[instance].set(sycl::coordinate_normalization_mode::unnormalized);
  // CHECK-NEXT: texDescInput[instance].set(sycl::addressing_mode::clamp);
  // CHECK-NEXT: texDescInput[instance].set(sycl::addressing_mode::clamp);
  memset(&resDescInput[instance], 0, sizeof(resDescInput[instance]));
  resDescInput[instance].resType = hipResourceTypeArray;
  resDescInput[instance].res.array.array = d_Input[instance];
  memset(&texDescInput[instance], 0, sizeof(texDescInput[instance]));
  texDescInput[instance].filterMode = hipFilterModeLinear;
  texDescInput[instance].normalizedCoords = false;
  texDescInput[instance].addressMode[0] = hipAddressModeBorder;
  texDescInput[instance].addressMode[1] = hipAddressModeBorder;

  // CHECK: if (tex_Input[instance] != NULL) {
  // CHECK-NEXT:   delete tex_Input[instance];
  // CHECK-NEXT:   tex_Input[instance] = NULL;
  // CHECK-NEXT: }
  // CHECK-NEXT: tex_Input[instance] = dpct::create_image_wrapper(resDescInput[instance], texDescInput[instance]);
  if (tex_Input[instance] != NULL) {
    hipDestroyTextureObject(tex_Input[instance]);
    tex_Input[instance] = NULL;
  }
  hipCreateTextureObject(&tex_Input[instance], &resDescInput[instance],
                          &texDescInput[instance], NULL);
}

// CHECK: void test_Kernel(dpct::image_accessor_ext<float, 2> tex_inArg, float *d_out,
// CHECK-NEXT: int yPitchOutInFloat,
// CHECK-NEXT: const sycl::nd_item<3> &item_ct1) {
__global__ void test_Kernel(hipTextureObject_t tex_inArg, float *d_out,
                            int yPitchOutInFloat) {
  // x and y are coordinates of the output 2D array
  int x = blockIdx.x * blockDim.x + threadIdx.x;
  int y = blockIdx.y;
  int i = y * yPitchOutInFloat + x;

  float rTextureCoord = x;
  float pTextureCoord = y;
  // CHECK: float V = tex_inArg.read(rTextureCoord + 0.5f, pTextureCoord + 0.5f);
  float V = tex2D<float>(tex_inArg, rTextureCoord + 0.5f, pTextureCoord + 0.5f);

  d_out[i] = V;
}

void test(float *d_Out, int rSize, int pSize, int pPitch) {
  int numThreadsPerBlock = 128;
  int blocks = (rSize + numThreadsPerBlock - 1) / numThreadsPerBlock;
  dim3 blockSz(numThreadsPerBlock);
  dim3 gridSz(blocks, pSize);
  //CHECK: dpct::get_in_order_queue().submit(
  //CHECK-NEXT:   [&](sycl::handler &cgh) {
  //CHECK-NEXT:     auto tex_Input_0_acc = static_cast<dpct::image_wrapper<float, 2> *>(tex_Input[0])->get_access(cgh);
  //CHECK-EMPTY:
  //CHECK-NEXT:     auto tex_Input_0_smpl = tex_Input[0]->get_sampler();
  //CHECK-EMPTY:
  //CHECK-NEXT:     int pPitch_sizeof_float_ct2 = pPitch / sizeof(float);
  //CHECK-EMPTY:
  //CHECK-NEXT:     cgh.parallel_for(
  //CHECK-NEXT:       sycl::nd_range<3>(gridSz * blockSz, blockSz),
  //CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:         test_Kernel(dpct::image_accessor_ext<float, 2>(tex_Input_0_smpl, tex_Input_0_acc), d_Out, pPitch_sizeof_float_ct2, item_ct1);
  //CHECK-NEXT:       });
  //CHECK-NEXT:   });
  test_Kernel<<<gridSz, blockSz>>>(tex_Input[0], d_Out, pPitch / sizeof(float));
}
#undef MAX_INSTANCES

