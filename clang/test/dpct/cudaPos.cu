// RUN: dpct --format-range=none -out-root %T/cudaPos %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cudaPos/cudaPos.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>

struct Bar {

  // CHECK: sycl::id<3> one{0, 0, 0}, two{0, 0, 0}, three{0, 0, 0};
  hipPos one, two, three;

  // CHECK: sycl::id<3> a{0, 0, 0};
  hipPos a{};

  // CHECK: sycl::id<3> b{1, 0, 0};
  hipPos b{1};

  // CHECK: sycl::id<3> c{1, 2, 0};
  hipPos c{1, 2};

  // CHECK: sycl::id<3> d{1, 2, 3};
  hipPos d{1, 2, 3};

  // CHECK: sycl::id<3> e = (sycl::id<3>{0, 0, 0});
  hipPos e = (hipPos());

  // CHECK: sycl::id<3> f = sycl::id<3>({1, 2, 3});
  hipPos f = hipPos({1, 2, 3});

  // CHECK: sycl::id<3> g = sycl::id<3>({1, 2, 0});
  hipPos g = hipPos({1, 2});

  // CHECK: Bar(sycl::id<3> a) {}
  Bar(hipPos a) {}

  // CHECK: Bar(size_t i) : a({i, 0, 0}), b{i, 0, 0}, c{0, 0, 0} {}
  Bar(size_t i) : a({i}), b{i}, c() {}

  Bar(size_t i, size_t j);

  // CHECK: Bar(size_t i, size_t j, size_t k) : a({i, j, k}), b{i, j, k}, c{0, 0, 0} {}
  Bar(size_t i, size_t j, size_t k) : a({i, j, k}), b{i, j, k}, c() {}
};

// CHECK: Bar::Bar(size_t i, size_t j) : a({i, j, 0}), b{i, j, 0}, c{0, 0, 0} {}
Bar::Bar(size_t i, size_t j) : a({i, j}), b{i, j}, c() {}

template <typename T> class A {};

// CHECK: template <typename T = sycl::id<3>> class B {};
template <typename T = hipPos> class B {};
template <typename... Ts> class C {};


int main() {
  // CHECK: sycl::id<3> a{0, 0, 0}, i{0, 0, 0}, j{0, 0, 0};
  hipPos a, i, j;

  // CHECK: sycl::id<3> b{1, 0, 0};
  hipPos b{1};

  // CHECK: sycl::id<3> c{1, 1, 0};
  hipPos c{1, 1};

  // CHECK: sycl::id<3> d{1, 1, 1};
  hipPos d{1, 1, 1};

  // CHECK: sycl::id<3> e{0, 0, 0};
  hipPos e{};

  // CHECK: sycl::id<3> f = sycl::id<3>{0, 0, 0};
  hipPos f = hipPos();

  // CHECK: sycl::id<3> g = sycl::id<3>{1, 2, 3};
  hipPos g = hipPos{1, 2, 3};

  // CHECK: sycl::id<3> h = (sycl::id<3>)sycl::id<3>{0, 0, 0};
  hipPos h = (hipPos)hipPos();

  // CHECK: sycl::id<3> k = sycl::id<3>(f);
  hipPos k = hipPos(f);

  // CHECK: sycl::id<3> m((sycl::id<3>{0, 0, 0}));
  hipPos m((hipPos()));

  // CHECK: A<sycl::id<3>> n;
  A<hipPos> n;

  // CHECK: B<> o;
  B<> o;

  // CHECK: C<sycl::id<3>, sycl::id<3>> p;
  C<hipPos, hipPos> p;

  // CHECK: sycl::id<3> *p1;
  hipPos *p1;

  // CHECK: sycl::id<3> **p2;
  hipPos **p2;

  // CHECK: sycl::id<3> &r1 = a;
  hipPos &r1 = a;

  // CHECK: sycl::id<3> &&rval = sycl::id<3>{0, 0, 0};
  hipPos &&rval = hipPos();
  return 0;
}
