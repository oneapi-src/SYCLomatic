//RUN: dpct -out-root %T/remove-all-headers %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
//RUN: FileCheck --input-file %T/remove-all-headers/remove-all-headers.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/remove-all-headers/remove-all-headers.dp.cpp -o %T/remove-all-headers/remove-all-headers.dp.o %}
//CHECK:#include <sycl/sycl.hpp>
//CHECK:#include <dpct/dpct.hpp>
//CHECK:#include <dpct/rng_utils.hpp>
#include <hip/hip_runtime.h>
#include <hiprand/hiprand.h>

