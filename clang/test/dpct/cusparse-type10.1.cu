// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none --out-root %T/cusparse-type10 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-type10/cusparse-type10.1.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

// HIPSPARSE_STATUS_NOT_SUPPORTED is available since v10.2.
int main(){
  //CHECK: int a6;
  //CHECK-NEXT: a6 = 10;
  hipsparseStatus_t a6;
  a6 = HIPSPARSE_STATUS_NOT_SUPPORTED;

  //CHECK:/*
  //CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:printf("Error string: %s", "hipsparseGetErrorString is not supported"/*hipsparseGetErrorString(status)*/);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:printf("Error name: %s", "hipsparseGetErrorName is not supported"/*hipsparseGetErrorName(status)*/);
  printf("Error string: %s", hipsparseGetErrorString(status));
  printf("Error name: %s", hipsparseGetErrorName(status));
}

