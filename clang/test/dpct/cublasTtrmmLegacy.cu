// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasTtrmmLegacy %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasTtrmmLegacy/cublasTtrmmLegacy.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublasTtrmmLegacy/cublasTtrmmLegacy.dp.cpp -o %T/cublasTtrmmLegacy/cublasTtrmmLegacy.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;
  int ldb = 275;
  const float *A_S = 0;
  float *B_S = 0;
  float alpha_S = 1.0f;
  const double *A_D = 0;
  double *B_D = 0;
  double alpha_D = 1.0;
  const hipComplex *A_C = 0;
  hipComplex *B_C = 0;
  hipComplex alpha_C = make_hipComplex(1.0f,0.0f);
  const hipDoubleComplex *A_Z = 0;
  hipDoubleComplex *B_Z = 0;
  hipDoubleComplex alpha_Z = make_hipDoubleComplex(1.0,0.0);


  //Legacy
  // CHECK: {
  // CHECK-NEXT: auto A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A_S);
  // CHECK-NEXT: auto B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B_S);
  // CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*dpct::blas::descriptor::get_saved_queue_ptr(), oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, alpha_S, A_S_buf_ct{{[0-9]+}}, lda, B_S_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasStrmm('L', 'U', 'N', 'N', m, n, alpha_S, A_S, lda, B_S, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(A_D);
  // CHECK-NEXT: auto B_D_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(B_D);
  // CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*dpct::blas::descriptor::get_saved_queue_ptr(), oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, alpha_D, A_D_buf_ct{{[0-9]+}}, lda, B_D_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasDtrmm('L', 'U', 'N', 'N', m, n, alpha_D, A_D, lda, B_D, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_C);
  // CHECK-NEXT: auto B_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_C);
  // CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*dpct::blas::descriptor::get_saved_queue_ptr(), oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, std::complex<float>(alpha_C.x(),alpha_C.y()), A_C_buf_ct{{[0-9]+}}, lda, B_C_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasCtrmm('L', 'U', 'N', 'N', m, n, alpha_C, A_C, lda, B_C, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_Z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_Z);
  // CHECK-NEXT: auto B_Z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_Z);
  // CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*dpct::blas::descriptor::get_saved_queue_ptr(), oneapi::mkl::side::left, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, std::complex<double>(alpha_Z.x(),alpha_Z.y()), A_Z_buf_ct{{[0-9]+}}, lda, B_Z_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT:}
  cublasZtrmm('L', 'U', 'N', 'N', m, n, alpha_Z, A_Z, lda, B_Z, ldb);
}

