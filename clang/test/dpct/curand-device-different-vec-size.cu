// RUN: dpct --format-range=none -extra-arg-before=-std=c++14 -out-root %T/curand-device-different-vec-size %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/curand-device-different-vec-size/curand-device-different-vec-size.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>
#include <cstdio>

const int WARP_SIZE = 32;
const int NBLOCKS = 640;
const int ITERATIONS = 1000000;


__global__ void picount(int *totals) {
  __shared__ int counter[WARP_SIZE];
  int tid = threadIdx.x + blockIdx.x * blockDim.x;

  // CHECK: dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>> rng;
  // CHECK: rng = dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>>(clock64(), 1234);
  hiprandState_t rng;
  hiprand_init(clock64(), tid, 1234, &rng);

  counter[threadIdx.x] = 0;

  for (int i = 0; i < ITERATIONS; i++) {
    //CHECK: float x = rng.generate<oneapi::mkl::rng::device::uniform<float>, 1>();
    //CHECK-NEXT: sycl::float2 y = rng.generate<oneapi::mkl::rng::device::gaussian<float>, 2>();
    float x = hiprand_uniform(&rng);
    float2 y = hiprand_normal2(&rng);
    counter[threadIdx.x] += 1 - int(x * x + y.x * y.x);
  }

  if (threadIdx.x == 0) {
    totals[blockIdx.x] = 0;
    for (int i = 0; i < WARP_SIZE; i++) {
      totals[blockIdx.x] += counter[i];
    }
  }
}


int main(int argc, char **argv) {
  int *dOut;
  picount<<<NBLOCKS, WARP_SIZE>>>(dOut);

  int size = 10;
  //CHECK: dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>> *RandomStates;
  hiprandState *RandomStates;
  //CHECK: RandomStates = (dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>> *)sycl::malloc_device(size * sizeof(dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>>) * 10, q_ct1);
  hipMalloc((void**)&RandomStates, size * sizeof(hiprandState) * 10);
  //CHECK: RandomStates = sycl::malloc_device<dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>>>(size, q_ct1);
  hipMalloc((void**)&RandomStates, size * sizeof(hiprandState));

  return 0;
}

