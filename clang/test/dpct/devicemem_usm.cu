// RUN: dpct --format-range=none -out-root %T/devicemem_usm %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/devicemem_usm/devicemem_usm.dp.cpp

#include <hip/hip_runtime.h>

#include <cassert>

#define NUM_ELEMENTS (/* Threads per block */ 16)

class TestStruct {
public:
  __device__ void test() {}
};

// CHECK: dpct::global_memory<TestStruct, 0> t1;
__device__ TestStruct t1;

// CHECK: __dpct_inline__ void member_acc(TestStruct &t1) {
// CHECK-NEXT:  t1.test();
// CHECK-NEXT:}
__global__ void member_acc() {
  t1.test();
}

// CHECK: dpct::global_memory<float, 1> in(NUM_ELEMENTS);
__device__ float in[NUM_ELEMENTS];
// CHECK: dpct::global_memory<int, 1> init(sycl::range<1>(4), {1, 2, 3, 4});
__device__ int init[4] = {1, 2, 3, 4};

// CHECK: __dpct_inline__ void kernel1(float *out, const sycl::nd_item<3> &[[ITEM:item_ct1]], float *in) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(2)] = in[{{.*}}[[ITEM]].get_local_id(2)];
// CHECK: }
__global__ void kernel1(float *out) {
  out[threadIdx.x] = in[threadIdx.x];
}

// CHECK: dpct::global_memory<int, 0> al;
__device__ int al;
// CHECK: dpct::global_memory<int, 0> ainit(NUM_ELEMENTS);
__device__ int ainit = NUM_ELEMENTS;

const int num_elements = 16;
// CHECK: dpct::global_memory<float, 1> fx(2);
// CHECK: dpct::global_memory<float, 2> fy(num_elements, 4 * num_elements);
__device__ float fx[2], fy[num_elements][4 * num_elements];

// CHECK: dpct::global_memory<float, 1> tmp(size);
const int size = 64;
__device__ float tmp[size];
// CHECK: __dpct_inline__ void kernel2(float *out, const sycl::nd_item<3> &[[ITEM:item_ct1]], int &al, float *fx,
// CHECK:              dpct::accessor<float, dpct::global, 2> fy, float *tmp) {
// CHECK:   out[{{.*}}[[ITEM]].get_local_id(2)] += al;
// CHECK:   fx[{{.*}}[[ITEM]].get_local_id(2)] = fy[{{.*}}[[ITEM]].get_local_id(2)][{{.*}}[[ITEM]].get_local_id(2)];
// CHECK:   tmp[1] = 1.0f;
// CHECK: }
__global__ void kernel2(float *out) {
  out[threadIdx.x] += al;
  fx[threadIdx.x] = fy[threadIdx.x][threadIdx.x];
  tmp[1] = 1.0f;
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  float h_in[NUM_ELEMENTS] = {0};
  float h_out[NUM_ELEMENTS] = {0};

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    h_in[i] = i;
    h_out[i] = -i;
  }

  const size_t array_size = sizeof(float) * NUM_ELEMENTS;

  hipMemcpyToSymbol(HIP_SYMBOL(in), h_in, array_size);

  const int h_a = 3;

  hipMemcpyToSymbol(HIP_SYMBOL(al), &h_a, sizeof(int));

  float *d_out = NULL;
  hipMalloc((void **)&d_out, array_size);

  const int threads_per_block = NUM_ELEMENTS;
  // CHECK:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       t1.init();
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto t1_ptr_ct1 = t1.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class member_acc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           member_acc(*t1_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  member_acc<<<1, threads_per_block>>>();
  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       in.init();
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto in_ptr_ct1 = in.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel1_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel1(d_out, item_ct1, in_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel1<<<1, threads_per_block>>>(d_out);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       al.init();
  // CHECK-NEXT:       fx.init();
  // CHECK-NEXT:       fy.init();
  // CHECK-NEXT:       tmp.init();
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto al_ptr_ct1 = al.get_ptr();
  // CHECK-NEXT:       auto fx_ptr_ct1 = fx.get_ptr();
  // CHECK-NEXT:       auto tmp_ptr_ct1 = tmp.get_ptr();
  // CHECK-EMPTY:
  // CHECK-NEXT:       auto fy_acc_ct1 = fy.get_access(cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel2_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, threads_per_block), sycl::range<3>(1, 1, threads_per_block)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel2(d_out, item_ct1, *al_ptr_ct1, fx_ptr_ct1, fy_acc_ct1, tmp_ptr_ct1);
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel2<<<1, threads_per_block>>>(d_out);

  hipMemcpy(h_out, d_out, array_size, hipMemcpyDeviceToHost);

  for (int i = 0; i < NUM_ELEMENTS; ++i) {
    assert(h_out[i] == i + h_a && "Value mis-calculated!");
  }

  return 0;
}

