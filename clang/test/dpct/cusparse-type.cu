// RUN: dpct --format-range=none --out-root %T/cusparse-type %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-type/cusparse-type.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusparse-type/cusparse-type.dp.cpp -o %T/cusparse-type/cusparse-type.dp.o %}
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int main(){
  //CHECK: oneapi::mkl::uplo a1;
  //CHECK-NEXT: a1 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: a1 = oneapi::mkl::uplo::upper;
  hipsparseFillMode_t a1;
  a1 = HIPSPARSE_FILL_MODE_LOWER;
  a1 = HIPSPARSE_FILL_MODE_UPPER;

  //CHECK: oneapi::mkl::diag a2;
  //CHECK-NEXT: a2 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: a2 = oneapi::mkl::diag::unit;
  hipsparseDiagType_t a2;
  a2 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  a2 = HIPSPARSE_DIAG_TYPE_UNIT;

  //CHECK: oneapi::mkl::index_base a3;
  //CHECK-NEXT: a3 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: a3 = oneapi::mkl::index_base::one;
  hipsparseIndexBase_t a3;
  a3 = HIPSPARSE_INDEX_BASE_ZERO;
  a3 = HIPSPARSE_INDEX_BASE_ONE;

  //CHECK: dpct::sparse::matrix_info::matrix_type a4;
  //CHECK-NEXT: a4 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: a4 = dpct::sparse::matrix_info::matrix_type::sy;
  //CHECK-NEXT: a4 = dpct::sparse::matrix_info::matrix_type::he;
  //CHECK-NEXT: a4 = dpct::sparse::matrix_info::matrix_type::tr;
  hipsparseMatrixType_t a4;
  a4 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  a4 = HIPSPARSE_MATRIX_TYPE_SYMMETRIC;
  a4 = HIPSPARSE_MATRIX_TYPE_HERMITIAN;
  a4 = HIPSPARSE_MATRIX_TYPE_TRIANGULAR;

  //CHECK: oneapi::mkl::transpose a5;
  //CHECK-NEXT: a5 = oneapi::mkl::transpose::nontrans;
  //CHECK-NEXT: a5 = oneapi::mkl::transpose::trans;
  //CHECK-NEXT: a5 = oneapi::mkl::transpose::conjtrans;
  hipsparseOperation_t a5;
  a5 = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  a5 = HIPSPARSE_OPERATION_TRANSPOSE;
  a5 = HIPSPARSE_OPERATION_CONJUGATE_TRANSPOSE;

  //CHECK: int a6;
  //CHECK-NEXT: a6 = 0;
  //CHECK-NEXT: a6 = 1;
  //CHECK-NEXT: a6 = 2;
  //CHECK-NEXT: a6 = 3;
  //CHECK-NEXT: a6 = 4;
  //CHECK-NEXT: a6 = 5;
  //CHECK-NEXT: a6 = 6;
  //CHECK-NEXT: a6 = 7;
  //CHECK-NEXT: a6 = 8;
  //CHECK-NEXT: a6 = 9;
  hipsparseStatus_t a6;
  a6 = HIPSPARSE_STATUS_SUCCESS;
  a6 = HIPSPARSE_STATUS_NOT_INITIALIZED;
  a6 = HIPSPARSE_STATUS_ALLOC_FAILED;
  a6 = HIPSPARSE_STATUS_INVALID_VALUE;
  a6 = HIPSPARSE_STATUS_ARCH_MISMATCH;
  a6 = HIPSPARSE_STATUS_MAPPING_ERROR;
  a6 = HIPSPARSE_STATUS_EXECUTION_FAILED;
  a6 = HIPSPARSE_STATUS_INTERNAL_ERROR;
  a6 = HIPSPARSE_STATUS_MATRIX_TYPE_NOT_SUPPORTED;
  a6 = HIPSPARSE_STATUS_ZERO_PIVOT;

#define VAL(x) NULL
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> a7;
  //CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrL=VAL(1);
  //CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrU=NULL;
  hipsparseMatDescr_t a7;
  hipsparseMatDescr_t descrL=VAL(1);
  hipsparseMatDescr_t descrU=NULL;

  //CHECK: sycl::queue* a8;
  hipsparseHandle_t a8;
}

//CHECK: void foo(oneapi::mkl::uplo a1,
//CHECK-NEXT:     oneapi::mkl::diag a2,
//CHECK-NEXT:     oneapi::mkl::index_base a3,
//CHECK-NEXT:     dpct::sparse::matrix_info::matrix_type a4,
//CHECK-NEXT:     oneapi::mkl::transpose a5,
//CHECK-NEXT:     int a6,
//CHECK-NEXT:     std::shared_ptr<dpct::sparse::matrix_info> a7,
//CHECK-NEXT:     sycl::queue* a8);
void foo(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8);

//CHECK:oneapi::mkl::uplo foo1();
//CHECK-NEXT:oneapi::mkl::diag foo2();
//CHECK-NEXT:oneapi::mkl::index_base foo3();
//CHECK-NEXT:dpct::sparse::matrix_info::matrix_type foo4();
//CHECK-NEXT:oneapi::mkl::transpose foo5();
//CHECK-NEXT:int foo6();
//CHECK-NEXT:std::shared_ptr<dpct::sparse::matrix_info> foo7();
//CHECK-NEXT:sycl::queue* foo8();
hipsparseFillMode_t foo1();
hipsparseDiagType_t foo2();
hipsparseIndexBase_t foo3();
hipsparseMatrixType_t foo4();
hipsparseOperation_t foo5();
hipsparseStatus_t foo6();
hipsparseMatDescr_t foo7();
hipsparseHandle_t foo8();

//CHECK:template<typename T>
//CHECK-NEXT:void bar1(oneapi::mkl::uplo a1,
//CHECK-NEXT:          oneapi::mkl::diag a2,
//CHECK-NEXT:          oneapi::mkl::index_base a3,
//CHECK-NEXT:          dpct::sparse::matrix_info::matrix_type a4,
//CHECK-NEXT:          oneapi::mkl::transpose a5,
//CHECK-NEXT:          int a6,
//CHECK-NEXT:          std::shared_ptr<dpct::sparse::matrix_info> a7,
//CHECK-NEXT:          sycl::queue* a8){}
template<typename T>
void bar1(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8){}

//CHECK:template<typename T>
//CHECK-NEXT:void bar2(oneapi::mkl::uplo a1,
//CHECK-NEXT:          oneapi::mkl::diag a2,
//CHECK-NEXT:          oneapi::mkl::index_base a3,
//CHECK-NEXT:          dpct::sparse::matrix_info::matrix_type a4,
//CHECK-NEXT:          oneapi::mkl::transpose a5,
//CHECK-NEXT:          int a6,
//CHECK-NEXT:          std::shared_ptr<dpct::sparse::matrix_info> a7,
//CHECK-NEXT:          sycl::queue* a8){}
template<typename T>
void bar2(hipsparseFillMode_t a1,
         hipsparseDiagType_t a2,
         hipsparseIndexBase_t a3,
         hipsparseMatrixType_t a4,
         hipsparseOperation_t a5,
         hipsparseStatus_t a6,
         hipsparseMatDescr_t a7,
         hipsparseHandle_t a8){}

// specialization
//CHECK:template<>
//CHECK-NEXT:void bar2<double>(oneapi::mkl::uplo a1,
//CHECK-NEXT:             oneapi::mkl::diag a2,
//CHECK-NEXT:             oneapi::mkl::index_base a3,
//CHECK-NEXT:             dpct::sparse::matrix_info::matrix_type a4,
//CHECK-NEXT:             oneapi::mkl::transpose a5,
//CHECK-NEXT:             int a6,
//CHECK-NEXT:             std::shared_ptr<dpct::sparse::matrix_info> a7,
//CHECK-NEXT:             sycl::queue* a8){}
template<>
void bar2<double>(hipsparseFillMode_t a1,
                  hipsparseDiagType_t a2,
                  hipsparseIndexBase_t a3,
                  hipsparseMatrixType_t a4,
                  hipsparseOperation_t a5,
                  hipsparseStatus_t a6,
                  hipsparseMatDescr_t a7,
                  hipsparseHandle_t a8){}


//CHECK: template void bar2<int>(oneapi::mkl::uplo a1,
//CHECK-NEXT:                   oneapi::mkl::diag a2,
//CHECK-NEXT:                   oneapi::mkl::index_base a3,
//CHECK-NEXT:                   dpct::sparse::matrix_info::matrix_type a4,
//CHECK-NEXT:                   oneapi::mkl::transpose a5,
//CHECK-NEXT:                   int a6,
//CHECK-NEXT:                   std::shared_ptr<dpct::sparse::matrix_info> a7,
//CHECK-NEXT:                   sycl::queue* a8);
template void bar2<int>(hipsparseFillMode_t a1,
                  hipsparseDiagType_t a2,
                  hipsparseIndexBase_t a3,
                  hipsparseMatrixType_t a4,
                  hipsparseOperation_t a5,
                  hipsparseStatus_t a6,
                  hipsparseMatDescr_t a7,
                  hipsparseHandle_t a8);

//CHECK: std::shared_ptr<dpct::sparse::matrix_info> b = 0, c = 0;
hipsparseMatDescr_t b = 0, c = 0;

