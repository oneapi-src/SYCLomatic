// FIXME
// UNSUPPORTED: system-windows
// RUN: dpct --no-dpcpp-extensions=device_info --format-range=none --usm-level=none -out-root %T/memory_management %s --cuda-include-path="%cuda-path/include" -output-file=memory_management_outputfile.txt -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_management/memory_management.dp.cpp %s

#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__constant__ float constData[123 * 4];

// CHECK: template<typename T>
//CHECK-NEXT: void test(){
//CHECK-NEXT:   int i = 0;
//CHECK-NEXT:   T** ptr;
//CHECK-NEXT:   T* array[10];
//CHECK-NEXT:   ptr[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT:   ptr[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT:   array[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT: }
template<typename T>
void test(){
  int i = 0;
  T** ptr;
  T* array[10];
  hipMalloc(&ptr[i], 10 * sizeof(T));
  hipMalloc(&(ptr[i]), 10 * sizeof(T));
  hipMalloc(&array[i], 10 * sizeof(T));
}

void fooo() {
  size_t size = 1234567 * sizeof(float);
  int id1, id2;
  float *h_A = (float *)malloc(size);
  hipPitchedPtr p_A;
  hipExtent e;
  float *d_A = NULL, *d_B = NULL;
  hipStream_t stream;
  hipMemcpy3DParms parms;
  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc((void **)&d_A, size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc(reinterpret_cast<void **>(&d_A), size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK: p_A = dpct::dpct_malloc(e);
  hipMalloc3D(&p_A, e);
  // CHECK: dpct::dpct_memset(d_A, 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK: dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK: dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);

  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size);
  hipMemsetAsync(d_A, 0xf, size);
  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size);
  hipMemsetAsync(d_A, 0xf, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size, *stream);
  hipMemsetAsync(d_A, 0xf, size, stream);

  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size, *stream);
  hipMemset2DAsync(d_A, size, 0xf, size, size, stream);

  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e, 0);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e, *stream);
  hipMemset3DAsync(p_A, 0xf, e, stream);

  // CHECK: dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(parms);
  hipMemcpy3D(&parms);

  struct hipMemcpy3DParms *parms_pointer;
  // Followed call can't be processed.
  hipMemcpy3D(parms_pointer);

  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host, *stream);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device, *stream);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host, *stream);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(parms);
  hipMemcpy3DAsync(&parms);
  // CHECK: dpct::async_dpct_memcpy(parms);
  hipMemcpy3DAsync(&parms, 0);
  // CHECK: dpct::async_dpct_memcpy(parms, *stream);
  hipMemcpy3DAsync(&parms, stream);

  // CHECK: dpct::dpct_memcpy(d_B, id1, d_A, id2, size);
  hipMemcpyPeer(d_B, id1, d_A, id2, size);
  // CHECK: /*
  // CHECK-NEXT: DPCT1124:{{[0-9]+}}: hipMemcpyPeerAsync is migrated to asynchronous memcpy API. While the origin API might be synchronous, it depends on the type of operand memory, so you may need to call wait() on event return by memcpy API to ensure synchronization behavior.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::async_dpct_memcpy(d_B, id1, d_A, id2, size, *stream);
  hipMemcpyPeerAsync(d_B, id1, d_A, id2, size, stream);

  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 0, hipMemcpyHostToDevice);
  // dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, 0);
  // dpct::async_dpct_memcpy(constData.get_ptr(stream), h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 0, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, 0);
  // dpct::async_dpct_memcpy(h_A, constData.get_ptr(*stream)), size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  if (1) {
    // CHECK:  return DPCT_CHECK_ERROR(*buffer = dpct::dpct_malloc(size));
    return hipMalloc(buffer, size);
  }
  if (1) {
    struct hipPitchedPtr pitch;
    struct hipExtent pitch_size;
    // CHECK:  return DPCT_CHECK_ERROR(pitch = dpct::dpct_malloc(pitch_size));
    return hipMalloc3D(&pitch, pitch_size);
  }
  if (1) {
    // CHECK:  return DPCT_CHECK_ERROR(*buffer = dpct::dpct_malloc(size, size, size));
    return hipMallocPitch(buffer, &size, size, size);
  }
}

void checkError(hipError_t err) {
}

void cuCheckError(hipError_t err) {
}
//CHECK: #define PITCH(a,b,c,d) a = (float *)dpct::dpct_malloc(b, c, d);
#define PITCH(a,b,c,d) hipMallocPitch(a, b, c, d);

void testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  float *d_B = NULL;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  // CHECK:  d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc((void **)&d_A, size);
  // CHECK:  dpct::err0 err = DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc(size));
  hipError_t err = hipMalloc((void **)&d_A, size);
  // CHECK:  checkError(DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc(size)));
  checkError(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)dpct::dpct_malloc(size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);

  // CHECK: PITCH((void **)&d_A, &size, size, size);
  PITCH((void **)&d_A, &size, size, size);
  int sz;
  // CHECK: d_A = (float *)dpct::dpct_malloc(*(size_t *)&size, size, size);
  hipMallocPitch((void **)&d_A, (size_t *)&size, size, size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(*(size_t *)&sz, size, size);
  hipMallocPitch((void **)&d_A, (size_t *)&sz, size, size);
  // CHECK:  err = DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc(size, size, size));
  err = hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK:  checkError(DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc(size, size, size)));
  checkError(hipMallocPitch((void **)&d_A, &size, size, size));

  // CHECK: p_A = dpct::dpct_malloc(e);
  hipMalloc3D(&p_A, e);
  // CHECK:  err = DPCT_CHECK_ERROR(p_A = dpct::dpct_malloc(e));
  err = hipMalloc3D(&p_A, e);
  // CHECK:  checkError(DPCT_CHECK_ERROR(p_A = dpct::dpct_malloc(e)));
  checkError(hipMalloc3D(&p_A, e));

  // CHECK:  dpct::dpct_memset(d_A, 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memset(d_A, 0xf, size));
  err = hipMemset(d_A, 0xf, size);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memset(d_A, 0xf, size)));
  checkError(hipMemset(d_A, 0xf, size));

  // CHECK:  dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memset(d_A, size, 0xf, size, size));
  err = hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memset(d_A, size, 0xf, size, size)));
  checkError(hipMemset2D(d_A, size, 0xf, size, size));

  // CHECK:  dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memset(p_A, 0xf, e));
  err = hipMemset3D(p_A, 0xf, e);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memset(p_A, 0xf, e)));
  checkError(hipMemset3D(p_A, 0xf, e));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host));
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host)));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device));
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device)));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host));
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host)));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device));
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device)));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic));
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic)));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host));
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host)));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device));
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device)));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device));
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device)));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host));
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host)));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic));
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic)));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy(parms);
  hipMemcpy3D(&parms);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(parms));
  err = hipMemcpy3D(&parms);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(parms)));
  checkError(hipMemcpy3D(&parms));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::device_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyDeviceToDevice));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::device_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), d_B, size));
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(constData.get_ptr(), d_B, size)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host));
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host)));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost));

  // CHECK:  dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host));
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host)));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device));
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device)));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice));


  // CHECK:  dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device));
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device)));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, constData.get_ptr(), size));
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, constData.get_ptr(), size)));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_free(d_A));
  err = hipFree(d_A);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_free(d_A)));
  checkError(hipFree(d_A));
  // CHECK:  free(h_A);
  free(h_A);
}

// CHECK: static dpct::global_memory<float, 1> d_A(1234567);
// CHECK: static dpct::global_memory<float, 1> d_B(1234567);
static __device__ float d_A[1234567];
static __device__ float d_B[1234567];

void testCommas_in_global_memory() {
  size_t size = 1234567 * sizeof(float);
  hipError_t err;
  float *h_A = (float *)malloc(size);

  // CHECK:  dpct::dpct_memset(d_A.get_ptr(), 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memset(d_A.get_ptr(), 0xf, size));
  err = hipMemset(d_A, 0xf, size);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memset(d_A.get_ptr(), 0xf, size)));
  checkError(hipMemset(d_A, 0xf, size));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host);
  hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host));
  err = hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host)));
  checkError(hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device));
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device)));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host));
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host)));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device);
  hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device));
  err = hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device)));
  checkError(hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic));
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic)));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device));
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic));
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size));
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size)));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host));
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host)));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device));
  err = hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device)));
  checkError(hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:   err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size));
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:   checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(h_A, d_B.get_ptr(), size)));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size));

  void *p_addr;
  // CHECK:  *(&p_addr) = d_A.get_ptr();
  hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK:  err = DPCT_CHECK_ERROR(*(&p_addr) = d_A.get_ptr());
  err = hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK:checkError(DPCT_CHECK_ERROR(*(&p_addr) = d_A.get_ptr()));
  checkError(hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A)));

  size_t size2;
  // CHECK: size2 = d_A.get_size();
  hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

  // CHECK:  err = DPCT_CHECK_ERROR(size2 = d_A.get_size());
  err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

  // CHECK:checkError(DPCT_CHECK_ERROR(size2 = d_A.get_size()));
  checkError(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

  size_t result1, result2;
  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: result2 = dpct::get_current_device().get_device_info().get_global_mem_size();
  hipMemGetInfo(&result1, &result2);

  hipError_t cu_err;
  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: cu_err = DPCT_CHECK_ERROR(result2 = dpct::get_current_device().get_device_info().get_global_mem_size());
  cu_err = hipMemGetInfo(&result1, &result2);

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: cuCheckError(DPCT_CHECK_ERROR(result2 = dpct::get_current_device().get_device_info().get_global_mem_size()));
  cuCheckError(hipMemGetInfo(&result1, &result2));

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: result2 = dpct::get_current_device().get_device_info().get_global_mem_size();
  hipMemGetInfo(&result1, &result2);


  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: err = DPCT_CHECK_ERROR(result2 = dpct::get_current_device().get_device_info().get_global_mem_size());
  err = hipMemGetInfo(&result1, &result2);

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: checkError(DPCT_CHECK_ERROR(result2 = dpct::get_current_device().get_device_info().get_global_mem_size()));
  checkError(hipMemGetInfo(&result1, &result2));

  hipDeviceptr_t  devicePtr;
  // CHECK: devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size);
  hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size);

  // CHECK: cu_err = DPCT_CHECK_ERROR(devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size));
  cu_err = hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size);
  // CHECK:  cuCheckError(DPCT_CHECK_ERROR(devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size)));
  cuCheckError(hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size));

  int* a;
  hipStream_t stream;
  int deviceID = 0;
  hipDevice_t cudevice =0;
  hipDeviceptr_t devPtr;
  // CHECK:/*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of hipMemPrefetchAsync is not supported.
  // CHECK-NEXT:*/
  hipMemPrefetchAsync (a, 100, deviceID, stream);

  // CHECK:/*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of hipMemPrefetchAsync is not supported.
  // CHECK-NEXT:*/
  hipMemPrefetchAsync(devPtr, 100, cudevice, stream);

  // CHECK:  free(h_A);
  free(h_A);
}

#define MY_CHECKER(CALL)                                                           \
    if ((CALL) != hipSuccess) { \
        exit(-1); \
    }

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}


template<typename T>
void uninstantiated_template_call(const T * d_data, size_t width, size_t height) {
  size_t datasize = width * height;
  T * data = new T[datasize];
  hipMemcpy3DParms parms;
  // CHECK:  assert_cuda(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host)));
  assert_cuda(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

#define DATAMACRO data+32*32

  // CHECK: dpct::dpct_memcpy(DATAMACRO, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(DATAMACRO, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(32*32+DATAMACRO, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(32*32+DATAMACRO, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host)));
  checkError(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: dpct::err0 err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host));
  hipError_t err = hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host)));
  MY_CHECKER(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host)));
  MY_ERROR_CHECKER(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: #define CUDAMEMCPY dpct::dpct_memcpy
  // CHECK-NEXT: CUDAMEMCPY(data, d_data, datasize * sizeof(T), dpct::device_to_host);
  #define CUDAMEMCPY hipMemcpy
  CUDAMEMCPY(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:  assert_cuda(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host)));
  assert_cuda(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK: dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(DATAMACRO, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(DATAMACRO, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(32*32+DATAMACRO, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(32*32+DATAMACRO, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host)));
  MY_CHECKER(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host)));
  MY_ERROR_CHECKER(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK: #define CUDAMEMCPY2D dpct::dpct_memcpy
  // CHECK-NEXT: CUDAMEMCPY2D(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  #define CUDAMEMCPY2D hipMemcpy2D
  CUDAMEMCPY2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(parms)));
  MY_CHECKER(hipMemcpy3D(&parms));

  // CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(dpct::dpct_memcpy(parms)));
  MY_ERROR_CHECKER(hipMemcpy3D(&parms));

  delete[] data;
}

#define N 1024
void test_segmentation_fault() {
  float *buffer;
  /*
  * Original code in getSizeString():
  * "SizeExpr->getBeginLoc()" cannot get the real SourceLocation of "N*sizeof(float)",
  * and results in boundary violation in "dpctGlobalInfo::getSourceManager().getCharacterData(SizeBegin)"
  * and fails with segmentation fault.
  */
  hipMalloc(&buffer, N*sizeof(float));
}

// CHECK: static dpct::global_memory<uint32_t, 1> d_error(1);
static __device__ uint32_t d_error[1];

void test_foo(){
  // CHECK: dpct::dpct_memset(d_error.get_ptr(), 0, sizeof(uint32_t));
  hipMemset(d_error, 0, sizeof(uint32_t));
}

void foobar() {
  int errorCode;

  hipChannelFormatDesc desc;
  hipExtent extent;
  unsigned int flags;
  hipArray_t array;

  // CHECK: desc = array->get_channel();
  // CHECK: extent = array->get_range();
  // CHECK: flags = 0;
  hipArrayGetInfo(&desc, &extent, &flags, array);

  //CHECK: checkError(DPCT_CHECK_ERROR([&](){
  //CHECK-NEXT:   desc = array->get_channel();
  //CHECK-NEXT:   extent = array->get_range();
  //CHECK-NEXT:   flags = 0;
  //CHECK-NEXT:   }()));
  checkError(hipArrayGetInfo(&desc, &extent, &flags, array));

  //CHECK: errorCode = DPCT_CHECK_ERROR([&](){
  //CHECK-NEXT:   desc = array->get_channel();
  //CHECK-NEXT:   extent = array->get_range();
  //CHECK-NEXT:   flags = 0;
  //CHECK-NEXT:   }());
  errorCode = hipArrayGetInfo(&desc, &extent, &flags, array);

  int host;
  // CHECK: flags = 0;
  hipHostGetFlags(&flags, &host);

  // CHECK: checkError(DPCT_CHECK_ERROR(flags = 0));
  checkError(hipHostGetFlags(&flags, &host));

  // CHECK: errorCode = DPCT_CHECK_ERROR(flags = 0);
  errorCode = hipHostGetFlags(&flags, &host);

  /*
  DPCT1082:{{[0-9]+}}: Migration of hipMemGenericAllocationHandle_t type is not supported.
  */
  hipMemGenericAllocationHandle_t a;
  /*
  DPCT1082:{{[0-9]+}}: Migration of hipMemAllocationProp type is not supported.
  */
  hipMemAllocationProp b;
  /*
  DPCT1082:{{[0-9]+}}: Migration of hipMemAccessDesc type is not supported.
  */
  hipMemAccessDesc c;


  int *devPtr;

  hipDeviceptr_t devicePtr;

  hipError_t cu_err;

  hipDeviceptr_t cuDevPtr;

  hipDevice_t cudevice =0;

  hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

  size_t count;
  // CHECK: int advice;
  hipMemoryAdvise advice;
  int device;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipMemAdvise(devPtr, count, advice, device);
  hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(hipMemAdvise(devPtr, count, advice, device));
  checkError(hipMemAdvise(devPtr, count, advice, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = hipMemAdvise(devPtr, count, advice, device);
  errorCode = hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipMemAdvise(devPtr, count, 0, device);
  hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(hipMemAdvise(devPtr, count, 0, device));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = hipMemAdvise(devPtr, count, 0, device);
  errorCode = hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);
}

// CHECK: void copy_dir_1 (dpct::memcpy_direction kind) {}
// CHECK-NEXT: void copy_dir_2 (dpct::memcpy_direction kind) {}
// CHECK-NEXT: void copy_dir_3 (dpct::memcpy_direction kind) {}
void copy_dir_1 (hipMemcpyKind kind) {}
void copy_dir_2 (enum hipMemcpyKind kind) {}
void copy_dir_3 (enum    hipMemcpyKind kind) {}

// CHECK: void copy_dir_1 (int kind) {}
// CHECK-NEXT: void copy_dir_2 (int kind) {}
// CHECK-NEXT: void copy_dir_3 (int kind) {}
void copy_dir_1 (hipComputeMode kind) {}
void copy_dir_2 (enum hipComputeMode kind) {}
void copy_dir_3 (enum    hipComputeMode kind) {}

