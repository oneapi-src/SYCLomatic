// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --no-dpcpp-extensions=device_info --format-range=none --usm-level=none -out-root %T/memory_management %s --cuda-include-path="%cuda-path/include" -output-file=memory_management_outputfile.txt -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_management/memory_management.dp.cpp %s

#include <hip/hip_runtime.h>
#include<hip/hip_runtime.h>

__constant__ float constData[123 * 4];

// CHECK: template<typename T>
//CHECK-NEXT: void test(){
//CHECK-NEXT:   int i = 0;
//CHECK-NEXT:   T** ptr;
//CHECK-NEXT:   T* array[10];
//CHECK-NEXT:   ptr[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT:   ptr[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT:   array[i] = (T *)dpct::dpct_malloc(10 * sizeof(T));
//CHECK-NEXT: }
template<typename T>
void test(){
  int i = 0;
  T** ptr;
  T* array[10];
  hipMalloc(&ptr[i], 10 * sizeof(T));
  hipMalloc(&(ptr[i]), 10 * sizeof(T));
  hipMalloc(&array[i], 10 * sizeof(T));
}

void fooo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  hipPitchedPtr p_A;
  hipExtent e;
  float *d_A = NULL;
  hipStream_t stream;
  hipMemcpy3DParms parms;
  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc((void **)&d_A, size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc(reinterpret_cast<void **>(&d_A), size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK: p_A = dpct::dpct_malloc(e);
  hipMalloc3D(&p_A, e);
  // CHECK: dpct::dpct_memset(d_A, 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK: dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK: dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);

  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size);
  hipMemsetAsync(d_A, 0xf, size);
  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size);
  hipMemsetAsync(d_A, 0xf, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, 0xf, size, *stream);
  hipMemsetAsync(d_A, 0xf, size, stream);

  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2DAsync(d_A, size, 0xf, size, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size, *stream);
  hipMemset2DAsync(d_A, size, 0xf, size, size, stream);

  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  hipMemset3DAsync(p_A, 0xf, e, 0);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e, *stream);
  hipMemset3DAsync(p_A, 0xf, e, stream);

  // CHECK: dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3D(&parms);

  struct hipMemcpy3DParms *parms_pointer;
  // Followed call can't be processed.
  hipMemcpy3D(parms_pointer);

  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, d_A, size, dpct::device_to_host, *stream);
  hipMemcpyAsync(h_A, d_A, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device, *stream);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, size, d_A, size, size, size, dpct::device_to_host, *stream);
  hipMemcpy2DAsync(h_A, size, d_A, size, size, size, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3DAsync(&parms, 0);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1, *stream);
  hipMemcpy3DAsync(&parms, stream);

  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 0, hipMemcpyHostToDevice);
  // dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, 0);
  // dpct::async_dpct_memcpy(constData.get_ptr(stream), h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 0, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, 0);
  // dpct::async_dpct_memcpy(h_A, constData.get_ptr(*stream)), size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost, stream);

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  free(h_A);
}

hipError_t mallocWrapper(void **buffer, size_t size) {
  if (1) {
    // CHECK:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:  return (*buffer = dpct::dpct_malloc(size), 0);
    return hipMalloc(buffer, size);
  }
  if (1) {
    struct hipPitchedPtr pitch;
    struct hipExtent pitch_size;
    // CHECK:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:  return (pitch = dpct::dpct_malloc(pitch_size), 0);
    return hipMalloc3D(&pitch, pitch_size);
  }
  if (1) {
    // CHECK:/*
    // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
    // CHECK-NEXT:*/
    // CHECK-NEXT:  return (*buffer = dpct::dpct_malloc(size, size, size), 0);
    return hipMallocPitch(buffer, &size, size, size);
  }
}

void checkError(hipError_t err) {
}

void cuCheckError(hipError_t err) {
}

void testCommas() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  float *d_B = NULL;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  // CHECK:  d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  dpct::err0 err = (d_A = (float *)dpct::dpct_malloc(size), 0);
  hipError_t err = hipMalloc((void **)&d_A, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((d_A = (float *)dpct::dpct_malloc(size), 0));
  checkError(hipMalloc((void **)&d_A, size));

  // CHECK: d_A = (float *)dpct::dpct_malloc(size, size, size);
  hipMallocPitch((void **)&d_A, &size, size, size);
  int sz;
  // CHECK: d_A = (float *)dpct::dpct_malloc(*((size_t *)&size), size, size);
  hipMallocPitch((void **)&d_A, (size_t *)&size, size, size);
  // CHECK: d_A = (float *)dpct::dpct_malloc(*((size_t *)&sz), size, size);
  hipMallocPitch((void **)&d_A, (size_t *)&sz, size, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (d_A = (float *)dpct::dpct_malloc(size, size, size), 0);
  err = hipMallocPitch((void **)&d_A, &size, size, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((d_A = (float *)dpct::dpct_malloc(size, size, size), 0));
  checkError(hipMallocPitch((void **)&d_A, &size, size, size));

  // CHECK: p_A = dpct::dpct_malloc(e);
  hipMalloc3D(&p_A, e);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (p_A = dpct::dpct_malloc(e), 0);
  err = hipMalloc3D(&p_A, e);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((p_A = dpct::dpct_malloc(e), 0));
  checkError(hipMalloc3D(&p_A, e));

  // CHECK:  dpct::dpct_memset(d_A, 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset(d_A, 0xf, size), 0);
  err = hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset(d_A, 0xf, size), 0));
  checkError(hipMemset(d_A, 0xf, size));

  // CHECK:  dpct::dpct_memset(d_A, size, 0xf, size, size);
  hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset(d_A, size, 0xf, size, size), 0);
  err = hipMemset2D(d_A, size, 0xf, size, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset(d_A, size, 0xf, size, size), 0));
  checkError(hipMemset2D(d_A, size, 0xf, size, size));

  // CHECK:  dpct::dpct_memset(p_A, 0xf, e);
  hipMemset3D(p_A, 0xf, e);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset(p_A, 0xf, e), 0);
  err = hipMemset3D(p_A, 0xf, e);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset(p_A, 0xf, e), 0));
  checkError(hipMemset3D(p_A, 0xf, e));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_host), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A, size, dpct::device_to_device), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A, size, dpct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host), 0);
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_host), 0));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToHost));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device), 0);
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_device), 0));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToDevice));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host), 0);
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::device_to_host), 0));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDeviceToHost));

  // CHECK:  dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic);
  hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic), 0);
  err = hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A, size, h_A, size, size, size, dpct::automatic), 0));
  checkError(hipMemcpy2D(d_A, size, h_A, size, size, size, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  hipMemcpy3D(&parms);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  err = hipMemcpy3D(&parms);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  checkError(hipMemcpy3D(&parms));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyHostToDevice));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::device_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), h_A, size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 0, hipMemcpyDeviceToDevice));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::device_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), d_B, size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 0, hipMemcpyDefault));

  // CHECK:  dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, d_B, size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size, 1, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:  dpct::dpct_memcpy(constData.get_ptr(), d_B, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(constData.get_ptr(), d_B, size), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(constData.get_ptr(), d_B, size), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(constData), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, constData.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToHost));

  // CHECK:  dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_B, constData.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 0, hipMemcpyDeviceToDevice));


  // CHECK:  dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:  dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_B, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_B, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_B, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:  dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(h_A, constData.get_ptr(), size), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(h_A, constData.get_ptr(), size), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size));

  // CHECK: dpct::dpct_free(d_A);
  hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_free(d_A), 0);
  err = hipFree(d_A);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_free(d_A), 0));
  checkError(hipFree(d_A));
  // CHECK:  free(h_A);
  free(h_A);
}

// CHECK: static dpct::global_memory<float, 1> d_A(1234567);
// CHECK: static dpct::global_memory<float, 1> d_B(1234567);
static __device__ float d_A[1234567];
static __device__ float d_B[1234567];

void testCommas_in_global_memory() {
  size_t size = 1234567 * sizeof(float);
  hipError_t err;
  float *h_A = (float *)malloc(size);

  // CHECK:  dpct::dpct_memset(d_A.get_ptr(), 0xf, size);
  hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memset(d_A.get_ptr(), 0xf, size), 0);
  err = hipMemset(d_A, 0xf, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memset(d_A.get_ptr(), 0xf, size), 0));
  checkError(hipMemset(d_A, 0xf, size));

  ///////// Host to host
  // CHECK:  dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host);
  hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host), 0);
  err = hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, h_A, size, dpct::host_to_host), 0));
  checkError(hipMemcpy(h_A, h_A, size, hipMemcpyHostToHost));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device), 0);
  err = hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device), 0));
  checkError(hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost));

  ///////// Device to Device
  // CHECK:  dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device);
  hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_B.get_ptr(), d_A.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpy(d_B, d_A, size, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic);
  hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic), 0);
  err = hipMemcpy(h_A, d_A, size, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::automatic), 0));
  checkError(hipMemcpy(h_A, d_A, size, hipMemcpyDefault));

  ///////// Host to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(d_A.get_ptr(), h_A, size, dpct::host_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), h_A, size, 0, hipMemcpyHostToDevice));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  err = hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(d_A), d_B, size, 0, hipMemcpyDeviceToDevice));

  ///////// Default
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(h_A, d_B.get_ptr(), size, dpct::automatic), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size, 0, hipMemcpyDefault));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size);
  hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(h_A, d_B.get_ptr(), size), 0);
  err = hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(h_A, d_B.get_ptr(), size), 0));
  checkError(hipMemcpyToSymbol(HIP_SYMBOL(h_A), d_B, size));

  ///////// Device to host
  // CHECK:  dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(h_A, d_A.get_ptr(), size, dpct::device_to_host), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_A), size, 0, hipMemcpyDeviceToHost));

  ///////// Device to device
  // CHECK:  dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device);
  hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0);
  err = hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(d_A.get_ptr(), d_B.get_ptr(), size, dpct::device_to_device), 0));
  checkError(hipMemcpyFromSymbol(d_A, HIP_SYMBOL(d_B), size, 0, hipMemcpyDeviceToDevice));

  ///////// Default parameter overload
  // CHECK:  dpct::dpct_memcpy(h_A, d_B.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   err = (dpct::dpct_memcpy(h_A, d_B.get_ptr(), size), 0);
  err = hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:   checkError((dpct::dpct_memcpy(h_A, d_B.get_ptr(), size), 0));
  checkError(hipMemcpyFromSymbol(h_A, HIP_SYMBOL(d_B), size));

  void *p_addr;
  // CHECK:  *(&p_addr) = d_A.get_ptr();
  hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK:  /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  err = (*(&p_addr) = d_A.get_ptr(), 0);
  err = hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A));

  // CHECK: /*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:checkError((*(&p_addr) = d_A.get_ptr(), 0));
  checkError(hipGetSymbolAddress(&p_addr, HIP_SYMBOL(d_A)));

  size_t size2;
  // CHECK: size2 = d_A.get_size();
  hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

  // CHECK: /*
  // CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  err = (size2 = d_A.get_size(), 0);
  err = hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));

  // CHECK: /*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:checkError((size2 = d_A.get_size(), 0));
  checkError(hipGetSymbolSize(&size2, HIP_SYMBOL(d_A)));

  size_t result1, result2;
  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: result2 = dpct::get_current_device().get_device_info().get_global_mem_size();
  hipMemGetInfo(&result1, &result2);

  hipError_t cu_err;
  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: cu_err = (result2 = dpct::get_current_device().get_device_info().get_global_mem_size(), 0);
  cu_err = hipMemGetInfo(&result1, &result2);

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: cuCheckError((result2 = dpct::get_current_device().get_device_info().get_global_mem_size(), 0));
  cuCheckError(hipMemGetInfo(&result1, &result2));

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: result2 = dpct::get_current_device().get_device_info().get_global_mem_size();
  hipMemGetInfo(&result1, &result2);


  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: err = (result2 = dpct::get_current_device().get_device_info().get_global_mem_size(), 0);
  err = hipMemGetInfo(&result1, &result2);

  // CHECK: /*
  // CHECK: DPCT1072:{{[0-9]+}}: SYCL currently does not support getting the available memory on the current device. You may need to adjust the code.
  // CHECK: */
  // CHECK: checkError((result2 = dpct::get_current_device().get_device_info().get_global_mem_size(), 0));
  checkError(hipMemGetInfo(&result1, &result2));

  hipDeviceptr_t  devicePtr;
  // CHECK: devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size);
  hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: cu_err = ((devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size), 0), 0);
  cu_err = hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  cuCheckError(((devicePtr = (dpct::device_ptr)dpct::dpct_malloc(size, size, size), 0), 0));
  cuCheckError(hipMemAllocPitch((hipDeviceptr_t *)&devicePtr, &size, size, size, size));

  int* a;
  hipStream_t stream;
  int deviceID = 0;
  hipDevice_t cudevice =0;
  hipDeviceptr_t devPtr;
  // CHECK:/*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of hipMemPrefetchAsync is not supported.
  // CHECK-NEXT:*/
  hipMemPrefetchAsync (a, 100, deviceID, stream);

  // CHECK:/*
  // CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of hipMemPrefetchAsync is not supported.
  // CHECK-NEXT:*/
  hipMemPrefetchAsync(devPtr, 100, cudevice, stream);

  // CHECK:  free(h_A);
  free(h_A);
}

#define MY_CHECKER(CALL)                                                           \
    if ((CALL) != hipSuccess) { \
        exit(-1); \
    }

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)
template <typename T>
void my_error_checker(T ReturnValue, char const *const FuncName) {}


template<typename T>
void uninstantiated_template_call(const T * d_data, size_t width, size_t height) {
  size_t datasize = width * height;
  T * data = new T[datasize];
  hipMemcpy3DParms parms;
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  assert_cuda((dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host), 0));
  assert_cuda(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

#define DATAMACRO data+32*32

  // CHECK: dpct::dpct_memcpy(DATAMACRO, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(DATAMACRO, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(32*32+DATAMACRO, d_data, datasize * sizeof(T), dpct::device_to_host);
  hipMemcpy(32*32+DATAMACRO, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host), 0));
  checkError(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: dpct::err0 err = (dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host), 0);
  hipError_t err = hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_CHECKER((dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host), 0));
  MY_CHECKER(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dpct_memcpy(data, d_data, datasize * sizeof(T), dpct::device_to_host), 0));
  MY_ERROR_CHECKER(hipMemcpy(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost));

  // CHECK: #define CUDAMEMCPY dpct::dpct_memcpy
  // CHECK-NEXT: CUDAMEMCPY(data, d_data, datasize * sizeof(T), dpct::device_to_host);
  #define CUDAMEMCPY hipMemcpy
  CUDAMEMCPY(data, d_data, datasize * sizeof(T), hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  assert_cuda((dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host), 0));
  assert_cuda(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK: dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(DATAMACRO, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(DATAMACRO, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(32*32+DATAMACRO, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  hipMemcpy2D(32*32+DATAMACRO, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_CHECKER((dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host), 0));
  MY_CHECKER(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dpct_memcpy(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host), 0));
  MY_ERROR_CHECKER(hipMemcpy2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost));

  // CHECK: #define CUDAMEMCPY2D dpct::dpct_memcpy
  // CHECK-NEXT: CUDAMEMCPY2D(data, datasize, d_data, datasize, datasize, datasize, dpct::device_to_host);
  #define CUDAMEMCPY2D hipMemcpy2D
  CUDAMEMCPY2D(data, datasize, d_data, datasize, datasize, datasize, hipMemcpyDeviceToHost);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_CHECKER((dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  MY_CHECKER(hipMemcpy3D(&parms));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT: MY_ERROR_CHECKER((dpct::dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  MY_ERROR_CHECKER(hipMemcpy3D(&parms));

  delete[] data;
}

#define N 1024
void test_segmentation_fault() {
  float *buffer;
  /*
  * Original code in getSizeString():
  * "SizeExpr->getBeginLoc()" cannot get the real SourceLocation of "N*sizeof(float)",
  * and results in boundary violation in "dpctGlobalInfo::getSourceManager().getCharacterData(SizeBegin)"
  * and fails with segmentation fault.
  */
  hipMalloc(&buffer, N*sizeof(float));
}

// CHECK: static dpct::global_memory<uint32_t, 1> d_error(1);
static __device__ uint32_t d_error[1];

void test_foo(){
  // CHECK: dpct::dpct_memset(d_error.get_ptr(), 0, sizeof(uint32_t));
  hipMemset(d_error, 0, sizeof(uint32_t));
}

void foobar() {
  int errorCode;

  hipChannelFormatDesc desc;
  hipExtent extent;
  unsigned int flags;
  hipArray_t array;

  // CHECK: desc = array->get_channel();
  // CHECK: extent = array->get_range();
  // CHECK: flags = 0;
  hipArrayGetInfo(&desc, &extent, &flags, array);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0));
  checkError(hipArrayGetInfo(&desc, &extent, &flags, array));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = ([&](){
  // CHECK-NEXT:   desc = array->get_channel();
  // CHECK-NEXT:   extent = array->get_range();
  // CHECK-NEXT:   flags = 0;
  // CHECK-NEXT:   }(), 0);
  errorCode = hipArrayGetInfo(&desc, &extent, &flags, array);

  int host;
  // CHECK: flags = 0;
  hipHostGetFlags(&flags, &host);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError((flags = 0, 0));
  checkError(hipHostGetFlags(&flags, &host));

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = (flags = 0, 0);
  errorCode = hipHostGetFlags(&flags, &host);

  int *devPtr;

  hipDeviceptr_t devicePtr;

  hipError_t cu_err;

  hipDeviceptr_t cuDevPtr;

  hipDevice_t cudevice =0;

  hipMemoryAdvise advise = hipMemAdviseUnsetPreferredLocation;

  size_t count;
  // CHECK: int advice;
  hipMemoryAdvise advice;
  int device;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipMemAdvise(devPtr, count, advice, device);
  hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(hipMemAdvise(devPtr, count, advice, device));
  checkError(hipMemAdvise(devPtr, count, advice, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = hipMemAdvise(devPtr, count, advice, device);
  errorCode = hipMemAdvise(devPtr, count, advice, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipMemAdvise(devPtr, count, 0, device);
  hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: checkError(hipMemAdvise(devPtr, count, 0, device));
  checkError(hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipMemAdvise is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1063:{{[0-9]+}}: Advice parameter is device-defined and was set to 0. You may need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: errorCode = hipMemAdvise(devPtr, count, 0, device);
  errorCode = hipMemAdvise(devPtr, count, hipMemAdviseSetReadMostly, device);
}

// CHECK: void copy_dir_1 (dpct::memcpy_direction kind) {}
// CHECK-NEXT: void copy_dir_2 (dpct::memcpy_direction kind) {}
// CHECK-NEXT: void copy_dir_3 (dpct::memcpy_direction kind) {}
void copy_dir_1 (hipMemcpyKind kind) {}
void copy_dir_2 (enum hipMemcpyKind kind) {}
void copy_dir_3 (enum    hipMemcpyKind kind) {}

// CHECK: void copy_dir_1 (int kind) {}
// CHECK-NEXT: void copy_dir_2 (int kind) {}
// CHECK-NEXT: void copy_dir_3 (int kind) {}
void copy_dir_1 (hipComputeMode kind) {}
void copy_dir_2 (enum hipComputeMode kind) {}
void copy_dir_3 (enum    hipComputeMode kind) {}

