// RUN: echo
// CHECK: #include <oneapi/dpl/execution>
// CHECK-NEXT: #include <oneapi/dpl/algorithm>
// CHECK-NEXT: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include "dnn.h"
// CHECK-NEXT: #include <dpct/dnnl_utils.hpp>
#include "dnn.h"

int test(hipdnnHandle_t handle) {
  hipdnnCreate(&handle);
  return 0;
}
