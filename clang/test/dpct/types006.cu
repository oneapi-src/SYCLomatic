// RUN: dpct -out-root %T/types006 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types006/types006.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/types006/types006.dp.cpp -o %T/types006/types006.dp.o %}


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

void foo() {
// CHECK: dpct::rng::device::rng_generator<oneapi::mkl::rng::device::philox4x32x10<1>> bar;
// CHECK-NEXT: auto lambda = [&bar] () {
// CHECK-NEXT:   return bar.generate<oneapi::mkl::rng::device::uniform<float>, 1>();
// CHECK-NEXT: };
  hiprandStatePhilox4_32_10_t bar;
  auto lambda = [&bar] __device__ () {
    return hiprand_uniform(&bar);
  };
}
