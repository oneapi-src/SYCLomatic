// RUN: dpct -out-root %T/types006 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types006/types006.dp.cpp


#include <hip/hip_runtime.h>
#include <hiprand/hiprand_kernel.h>

void foo() {
// CHECK: dpct::rng::device::rng_generator<oneapi::mkl::rng::device::philox4x32x10<1>> bar;
// CHECK-NEXT: auto lambda = [&bar] () {
// CHECK-NEXT:   return bar.generate<oneapi::mkl::rng::device::uniform<float>, 1>();
// CHECK-NEXT: };
  hiprandStatePhilox4_32_10_t bar;
  auto lambda = [&bar] __device__ () {
    return hiprand_uniform(&bar);
  };
}
