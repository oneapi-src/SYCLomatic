// RUN: dpct --format-range=none -out-root %T/module_main %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/module_main/module_main.dp.cpp

//CHECK: #include <dpct/dpct.hpp>

#include "hip/hip_runtime.h"
#include <string>
int main(){
    //CHECK: dpct::kernel_library M;
    hipModule_t M;
    //CHECK: dpct::kernel_function F;
    hipFunction_t F;
    std::string Path, FunctionName, Data;
    //CHECK: /*
    //CHECK-NEXT: DPCT1103:{{[0-9]+}}: 'Path.c_str()' should be a dynamic library. The dynamic library should supply wrapped kernel functions.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = dpct::load_kernel_library(Path.c_str());
    hipModuleLoad(&M, Path.c_str());
    //CHECK: /*
    //CHECK-NEXT: DPCT1104:{{[0-9]+}}: 'Data.c_str()' should point to a dynamic library loaded in memory. The dynamic library should supply wrapped kernel functions.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = dpct::load_kernel_library_mem(Data.c_str());
    hipModuleLoadData(&M, Data.c_str());

    //CHECK: /*
    //CHECK-NEXT: DPCT1104:{{[0-9]+}}: 'Data.c_str()' should point to a dynamic library loaded in memory. The dynamic library should supply wrapped kernel functions.
    //CHECK-NEXT: */
    //CHECK-NEXT: M = dpct::load_kernel_library_mem(Data.c_str());
    hipModuleLoadDataEx(&M, Data.c_str(), 0, NULL, NULL);

    //CHECK: F = dpct::get_kernel_function(M, FunctionName.c_str());
    hipModuleGetFunction(&F, M, FunctionName.c_str());


    int    *argBuffer[3];
    size_t  argBufferSize = sizeof(argBuffer);
    //CHECK: void *extra[] = {((void *) 2), &argBufferSize,
    //CHECK-NEXT: ((void *) 1), argBuffer,
    //CHECK-NEXT: ((void *) 0)};
    void   *extra[] = {HIP_LAUNCH_PARAM_BUFFER_SIZE, &argBufferSize,
                       HIP_LAUNCH_PARAM_BUFFER_POINTER, argBuffer,
                       HIP_LAUNCH_PARAM_END};

    int sharedSize;
    hipStream_t s;
    void **param;
    //CHECK:  dpct::invoke_kernel_function(F, *s, sycl::range<3>(32, 16, 1), sycl::range<3>(64, 32, 4), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, s, param, extra);
    //CHECK:  dpct::invoke_kernel_function(F, q_ct1, sycl::range<3>(32, 16, 1), sycl::range<3>(64, 32, 4), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, 0, param, extra);
    //CHECK:  dpct::invoke_kernel_function(F, q_ct1, sycl::range<3>(32, 16, 1), sycl::range<3>(64, 32, 4), sharedSize, param, extra);
    hipModuleLaunchKernel(F, 1, 16, 32, 4, 32, 64, sharedSize, hipStreamLegacy, param, extra);

    //CHECK: dpct::image_wrapper_base_p tex;
    //CHECK: tex = dpct::get_image_wrapper(M, "tex");
    hipTexRef tex;
    hipModuleGetTexRef(&tex, M, "tex");

    //CHECK: dpct::unload_kernel_library(M);
    hipModuleUnload(M);

    //CHECK: if (DPCT_CHECK_ERROR(dpct::unload_kernel_library(M))==0) {
    if (hipModuleUnload(M)==0) {
      printf("unload failed\n");
    }

    return 0;
}