// RUN: dpct --enable-profiling=0 --format-range=none --usm-level=none -out-root %T/cuda-event-api %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cuda-event-api/cuda-event-api.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cuda-event-api/cuda-event-api.dp.cpp -o %T/cuda-event-api/cuda-event-api.dp.o %}


#include <hip/hip_runtime.h>
#include <stdio.h>

template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

//CHECK: #define CudaEvent(X) X = new sycl::event()
#define CudaEvent(X) hipEventCreate(&X)

#define MY_CHECKER(CALL) do {                           \
  hipError_t Error = CALL;                             \
  if (Error != hipSuccess) {                           \
  }                                                     \
} while(0)

__global__ void kernelFunc()
{
}

int main(int argc, char* argv[]) {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.out_of_order_queue();
  // CHECK: dpct::event_ptr start, stop;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
  // CHECK-EMPTY:
  // CHECK: start = new sycl::event();
  // CHECK: stop = new sycl::event();
  // CHECK-EMPTY:
  // CHECK-NEXT: float elapsed_time;
  // CHECK-EMPTY:
  // CHECK-NEXT: dev_ct1.queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK-NEXT: int blocks = 32, threads = 32;
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // CHECK: printf("<<<\n");
  // CHECK-NEXT: CudaEvent(start);
  // CHECK-NEXT: printf(">>>\n");
  printf("<<<\n");
  CudaEvent(start);
  printf(">>>\n");


  // CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(start = new sycl::event()));
  // CHECK: dpct::err0 et = DPCT_CHECK_ERROR(stop = new sycl::event());
  MY_ERROR_CHECKER(hipEventCreate(&start));
  hipError_t et = hipEventCreate(&stop);


  // kernel call without sync
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: start_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(start, 0);

  // kernel call without sync
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: start_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: *start = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord(start, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-f]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: start_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(start, 0));

  // CHECK: if (0)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   start_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

  // kernel call with sync
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(stop, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));

  // CHECK: if (1)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  if (1)
    MY_ERROR_CHECKER(hipEventRecord(stop, 0));

  // kernel call without sync
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[a-f0-9]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(stop, 0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT: */
  // CHECK-NEXT: stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier()));
  MY_ERROR_CHECKER(hipEventRecord(stop, 0));

  // CHECK: if (0)
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1012:{{[0-9a-z]+}}: Detected kernel execution time measurement pattern and generated an initial code for time measurements in SYCL. You can change the way time is measured depending on your goals.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   /*
  // CHECK-NEXT:   DPCT1024:{{[0-9a-f]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
  // CHECK-NEXT:   */
  // CHECK-NEXT:   start_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:   MY_ERROR_CHECKER(DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier()));
  if (0)
    MY_ERROR_CHECKER(hipEventRecord(start, 0));

  // CHECK:  MY_ERROR_CHECKER(0);
  MY_ERROR_CHECKER(hipEventRecord(start));

  // kernel call without sync
  // CHECK:  DPCT1049:{{[0-9a-f]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
  // CHECK-NEXT:  */
  // CHECK-NEXT:  q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:        sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:        [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:          kernelFunc();
  // CHECK-NEXT:        });
  kernelFunc<<<blocks,threads>>>();

  // CHECK:  dpct::get_current_device().queues_wait_and_throw();
  // CHECK-NEXT:  stop_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT:  elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
  hipEventRecord(stop, 0);
  hipEventSynchronize(stop);
  hipEventElapsedTime(&elapsed_time, start, stop);

  // CHECK: MY_ERROR_CHECKER(DPCT_CHECK_ERROR((elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count())));
  MY_ERROR_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

  // kernel call without sync
  // CHECK:   q_ct1.parallel_for<dpct_kernel_name<class kernelFunc_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, blocks) * sycl::range<3>(1, 1, threads), sycl::range<3>(1, 1, threads)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernelFunc();
  // CHECK-NEXT:         });
  kernelFunc<<<blocks,threads>>>();

  hipEventCreate(&stop);

  // CHECK: dpct::err0 e = (int)stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT: MY_ERROR_CHECKER(e);
  // CHECK-NEXT: MY_ERROR_CHECKER((int)stop->get_info<sycl::info::event::command_execution_status>());
  // CHECK-NEXT: if (0 == (int)stop->get_info<sycl::info::event::command_execution_status>()){}
  // CHECK-NEXT: while((int)stop->get_info<sycl::info::event::command_execution_status>() != 0){}
  // CHECK-NEXT: for(;0 != (int)stop->get_info<sycl::info::event::command_execution_status>();){}
  // CHECK-NEXT: do{}while((int)stop->get_info<sycl::info::event::command_execution_status>() == 0);
  // CHECK-NEXT: {
  // CHECK-NEXT:   int *a;
  // CHECK-NEXT:   sycl::info::event_command_status e1;
  // CHECK-NEXT:   e1 = stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT:   if (sycl::info::event_command_status::complete != stop->get_info<sycl::info::event::command_execution_status>()) {}
  // CHECK-NEXT:   if (e1 == sycl::info::event_command_status::complete){}
  // CHECK-NEXT:   while(e1 != sycl::info::event_command_status::complete) {
  // CHECK-NEXT:     e1 = stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT:   }
  // CHECK-NEXT:   for(;e1 != sycl::info::event_command_status::complete;){
  // CHECK-NEXT:     e1 = stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT:   }
  // CHECK-NEXT:   {
  // CHECK-NEXT:     dpct::err0 e;
  // CHECK-NEXT:     e = (int)stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT:     e = DPCT_CHECK_ERROR(a = (int *)dpct::dpct_malloc(sizeof(int)));
  // CHECK-NEXT:   }
  // CHECK-NEXT:   dpct::err0 et1, et2;
  // CHECK-NEXT:   et1 = (int)stop->get_info<sycl::info::event::command_execution_status>();
  // CHECK-NEXT:   et2 = DPCT_CHECK_ERROR(a = (int *)dpct::dpct_malloc(sizeof(int)));
  // CHECK-NEXT: }

  hipError_t e = hipEventQuery(stop);
  MY_ERROR_CHECKER(e);
  MY_ERROR_CHECKER(hipEventQuery(stop));
  if (hipErrorNotReady != hipEventQuery(stop)){}
  while(hipEventQuery(stop) == hipErrorNotReady){}
  for(;hipErrorNotReady == hipEventQuery(stop);){}
  do{}while(hipEventQuery(stop) != hipErrorNotReady);
  {
    int *a;
    hipError_t e1;
    e1 = hipEventQuery(stop);
    if (hipSuccess != hipEventQuery(stop)) {}
    if (e1 == hipSuccess){}
    while(e1 != hipSuccess) {
      e1 = hipEventQuery(stop);
    }
    for(;e1 != hipSuccess;){
      e1 = hipEventQuery(stop);
    }
    {
      hipError_t e;
      e = hipEventQuery(stop);
      e = hipMalloc(&a, sizeof(int));
    }
    hipError_t et1, et2;
    et1 = hipEventQuery(stop);
    et2 = hipMalloc(&a, sizeof(int));
  }

  // CHECK: dev_ct1.queues_wait_and_throw();
  // CHECK-EMPTY:
  // CHECK: et = DPCT_CHECK_ERROR(dpct::destroy_event(stop));
  // CHECK-NEXT: }
  hipDeviceSynchronize();

  hipEventDestroy(start);
  hipEventDestroy(stop);
  MY_ERROR_CHECKER(hipEventDestroy(start));
  et = hipEventDestroy(stop);
}

void foo() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // CHECK: start_ct1 = std::chrono::steady_clock::now();
  // CHECK: MY_CHECKER(0);
  MY_CHECKER(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
  // CHECK: stop_ct1 = std::chrono::steady_clock::now();
  // CHECK: MY_CHECKER(0);
  MY_CHECKER(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);

  // CHECK: MY_CHECKER(DPCT_CHECK_ERROR((elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count())));
  MY_CHECKER(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);

  {
    // CHECK: start_ct1 = std::chrono::steady_clock::now();
    // CHECK-NEXT: dpct::err0 err = DPCT_CHECK_ERROR(*start = q_ct1.ext_oneapi_submit_barrier());
    hipError_t err = hipEventRecord(start, 0);
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    // CHECK-NEXT: err = DPCT_CHECK_ERROR(*stop = q_ct1.ext_oneapi_submit_barrier());
    err = hipEventRecord(stop, 0);
    if (hipSuccess != err) {
      printf("%s\n", hipGetErrorString( err));
    }
  }
}

void fun(int) {}

void bar() {
  hipEvent_t start, stop;

  hipEventCreate(&start);
  hipEventCreate(&stop);

  float elapsed_time;

  hipDeviceSynchronize();

  int blocks = 32, threads = 32;

  // CHECK: start_ct1 = std::chrono::steady_clock::now();
  // CHECK-NEXT: fun(0);
  fun(hipEventRecord(start, 0));
  kernelFunc<<<blocks,threads>>>();
  // CHECK: stop_ct1 = std::chrono::steady_clock::now();
  // CHECK: fun(0);
  fun(hipEventRecord(stop, 0));

  hipEventSynchronize(stop);
  // CHECK: fun(DPCT_CHECK_ERROR((elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count())));
  fun(hipEventElapsedTime(&elapsed_time, start, stop));

  hipEventDestroy(start);
  hipEventDestroy(stop);
}

struct Node {
 // CHECK: dpct::event_ptr start;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
 hipEvent_t start;
 // CHECK: dpct::event_ptr end;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> end_ct1;
 hipEvent_t end;
 // CHECK: dpct::event_ptr *ev[100];
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> ev_ct1[100];
 hipEvent_t *ev[100];
 // CHECK: dpct::event_ptr events[100];
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1[100];
 hipEvent_t events[100];
 // CHECK: dpct::event_ptr *p_events;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> p_events_ct1_0;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> p_events_ct1_1;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> p_events_ct1_2;
 // CHECK: std::chrono::time_point<std::chrono::steady_clock> p_events_ct1_3;
 hipEvent_t *p_events;
};

void foo2(Node *n) {
  float elapsed_time;

  // CHECK: n->start_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(n->start, 0);
  // CHECK: n->start_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(n->start, 0);
  // do something
  // CHECK: n->end_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(n->end, 0);
  // CHECK: n->end_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(n->end, 0);
  // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(n->end_ct1 - n->start_ct1).count();
  hipEventElapsedTime(&elapsed_time, n->start, n->end);
  {
    int errorCode;
    // CHECK: n->start_ct1 = std::chrono::steady_clock::now();
    // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(*n->start = q_ct1.ext_oneapi_submit_barrier()));
    MY_CHECKER(hipEventRecord(n->start, 0));
    // CHECK: n->start_ct1 = std::chrono::steady_clock::now();
    // CHECK: errorCode = DPCT_CHECK_ERROR(*n->start = q_ct1.ext_oneapi_submit_barrier());
    errorCode = hipEventRecord(n->start, 0);
  }

  Node node;
  // CHECK: node.start_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(node.start, 0);
  // CHECK: node.start_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(node.start, 0);
  // do something
  // CHECK: node.end_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(node.end, 0);
  // CHECK: node.end_ct1 = std::chrono::steady_clock::now();
  hipEventRecord(node.end, 0);
  // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(node.end_ct1 - node.start_ct1).count();
  hipEventElapsedTime(&elapsed_time, node.start, node.end);
  {
    int errorCode;
    // CHECK: node.start_ct1 = std::chrono::steady_clock::now();
    // CHECK: MY_CHECKER(DPCT_CHECK_ERROR(*node.start = q_ct1.ext_oneapi_submit_barrier()));
    MY_CHECKER(hipEventRecord(node.start, 0));
    // CHECK: node.start_ct1 = std::chrono::steady_clock::now();
    // CHECK: errorCode = DPCT_CHECK_ERROR(*node.start = q_ct1.ext_oneapi_submit_barrier());
    errorCode = hipEventRecord(node.start, 0);
  }

  {
    // CHECK: node.events_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(node.events[0]);
    // CHECK: node.events_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(node.events[0]);
    // CHECK: node.events_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(node.events[23]);
    // CHECK: node.events_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(node.events[23]);
    // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(node.events_ct1[23] - node.events_ct1[0]).count();
    hipEventElapsedTime(&elapsed_time, node.events[0], node.events[23]);
  }

  {
    // CHECK: node.ev_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(*node.ev[0]);
    // CHECK: node.ev_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(*node.ev[0]);
    // CHECK: node.ev_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(*node.ev[23]);
    // CHECK: node.ev_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(*node.ev[23]);
    // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(node.ev_ct1[23] - node.ev_ct1[0]).count();
    hipEventElapsedTime(&elapsed_time, *node.ev[0], *node.ev[23]);
  }

  {
    // CHECK: (&node)->ev_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(*(&node)->ev[0]);
    // CHECK: (&node)->ev_ct1[0] = std::chrono::steady_clock::now();
    hipEventRecord(*(&node)->ev[0]);
    // CHECK: (&node)->ev_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(*(&node)->ev[23]);
    // CHECK: (&node)->ev_ct1[23] = std::chrono::steady_clock::now();
    hipEventRecord(*(&node)->ev[23]);
    // CHECK: elapsed_time = std::chrono::duration<float, std::milli>((&node)->ev_ct1[23] - (&node)->ev_ct1[0]).count();
    hipEventElapsedTime(&elapsed_time, *(&node)->ev[0], *(&node)->ev[23]);
  }

  {
    // CHECK: n->p_events_ct1_0 = std::chrono::steady_clock::now();
    hipEventRecord(n->p_events[0]);
    // CHECK: n->p_events_ct1_1 = std::chrono::steady_clock::now();
    hipEventRecord(n->p_events[1]);
    // CHECK: n->p_events_ct1_2 = std::chrono::steady_clock::now();
    hipEventRecord(n->p_events[2]);
    // CHECK: n->p_events_ct1_3 = std::chrono::steady_clock::now();
    hipEventRecord(n->p_events[3]);
  }
}

class C {
  // CHECK: dpct::event_ptr start, stop;
  // CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> start_ct1;
  // CHECK-NEXT: std::chrono::time_point<std::chrono::steady_clock> stop_ct1;
  hipEvent_t start, stop;
  float elapsed_time;
  void a() {
    // CHECK: start_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(start, 0);
  }
  void b() {
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(stop, 0);
    // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventElapsedTime(&elapsed_time, start, stop);
  }
  void c() {
    hipEventRecord(start, 0);
  }
  void d() {
    // CHECK: stop_ct1 = std::chrono::steady_clock::now();
    hipEventRecord(stop, 0);
    // CHECK: elapsed_time = std::chrono::duration<float, std::milli>(stop_ct1 - start_ct1).count();
    hipEventElapsedTime(&elapsed_time, start, stop);
  }
};

struct S {
  hipEvent_t *events;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_0;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_1;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_2;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_3;
};

void foo(int n) {
  // CHECK: dpct::event_ptr *events = new dpct::event_ptr[n];
  hipEvent_t *events = new hipEvent_t[n];
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_0;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_1;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_2;
  // CHECK: std::chrono::time_point<std::chrono::steady_clock> events_ct1_3;

  // CHECK: events_ct1_0 = std::chrono::steady_clock::now();
  hipEventRecord(events[0]);
  // CHECK: events_ct1_1 = std::chrono::steady_clock::now();
  hipEventRecord(events[1]);
  // CHECK: events_ct1_2 = std::chrono::steady_clock::now();
  hipEventRecord(events[2]);
  // CHECK: events_ct1_3 = std::chrono::steady_clock::now();
  hipEventRecord(events[3]);

  S s;
  // CHECK: s.events_ct1_0 = std::chrono::steady_clock::now();
  hipEventRecord(s.events[0]);
  // CHECK: s.events_ct1_1 = std::chrono::steady_clock::now();
  hipEventRecord(s.events[1]);
  // CHECK: s.events_ct1_2 = std::chrono::steady_clock::now();
  hipEventRecord(s.events[2]);
  // CHECK: s.events_ct1_3 = std::chrono::steady_clock::now();
  hipEventRecord(s.events[3]);

  S *s2 = new S;
  // CHECK: s2->events_ct1_0 = std::chrono::steady_clock::now();
  hipEventRecord(s2->events[0]);
  // CHECK: s2->events_ct1_1 = std::chrono::steady_clock::now();
  hipEventRecord(s2->events[1]);
  // CHECK: s2->events_ct1_2 = std::chrono::steady_clock::now();
  hipEventRecord(s2->events[2]);
  // CHECK: s2->events_ct1_3 = std::chrono::steady_clock::now();
  hipEventRecord(s2->events[3]);
}

void barr(int maxCalls) {
  hipEvent_t evtStart[maxCalls];
  hipEvent_t evtEnd[maxCalls];
  float time[maxCalls];
  for (int i = 0; i < maxCalls; i++) {
    hipEventCreate( &(evtStart[i]) );
    hipEventCreate( &(evtEnd[i]) );
    time[i] = 0.0;
  }

  // CHECK: evtStart_ct1[0] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[0], 0 );
  // CHECK: evtEnd[0]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[0] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[0], 0 );

  // CHECK: evtStart_ct1[1] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[1], 0 );
  // CHECK: evtEnd[1]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[1] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[1], 0 );

  // CHECK: evtStart_ct1[2] = std::chrono::steady_clock::now();
  hipEventRecord( evtStart[2], 0 );
  // CHECK: evtEnd[2]->wait();
  kernelFunc<<<1, 1>>>();
  // CHECK: evtEnd_ct1[2] = std::chrono::steady_clock::now();
  hipEventRecord( evtEnd[2], 0 );

  // CHECK: dev_ct1.queues_wait_and_throw();
  hipDeviceSynchronize();

  float total;
  int i=0;
  hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
  float timesum = 0.0f;
  for (int i = 1; i < maxCalls; i++) {
    hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
    timesum += time[i];
  }
  hipEventElapsedTime( &total, evtStart[1], evtEnd[maxCalls-1]);
}

