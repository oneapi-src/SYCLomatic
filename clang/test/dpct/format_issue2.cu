
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T/format_issue2 %s --cuda-include-path="%cuda-path/include" --  -x cuda --cuda-host-only -DCOMPILATION
// RUN: FileCheck %s --match-full-lines --input-file %T/format_issue2/format_issue2.dp.cpp

float a;int
#define AAA
#define BBB
#define CCC
#define DDD
#define EEE
#define FFF
foo1(){ return 1; }
int main(){ int2 i; }


// CHECK: float a;int
// CHECK-NEXT: #define AAA
// CHECK-NEXT: #define BBB
// CHECK-NEXT: #define CCC
// CHECK-NEXT: #define DDD
// CHECK-NEXT: #define EEE
// CHECK-NEXT: #define FFF
// CHECK-NEXT: foo1(){ return 1; }
// CHECK-NEXT: int main() { sycl::mint2 i; }