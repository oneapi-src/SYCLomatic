// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1
// RUN: dpct --format-range=none --out-root %T/cusparse-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int foo(int aaaaa){
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> descr1 = 0, descr2 = 0;
  //CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descr3 = 0;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: dpct::sparse::matrix_info::matrix_type type0 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: descrA->set_diag((oneapi::mkl::diag)aaaaa);
  //CHECK-NEXT: descrA->set_uplo((oneapi::mkl::uplo)aaaaa);
  //CHECK-NEXT: descrA->set_index_base((oneapi::mkl::index_base)aaaaa);
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)aaaaa);
  //CHECK-NEXT: diag0 = descrA->get_diag();
  //CHECK-NEXT: fill0 = descrA->get_uplo();
  //CHECK-NEXT: base0 = descrA->get_index_base();
  //CHECK-NEXT: type0 = descrA->get_matrix_type();
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = std::make_shared<dpct::sparse::matrix_info>();
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)aaaaa);
  //CHECK-NEXT: descrA->set_index_base(oneapi::mkl::index_base::zero);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, (oneapi::mkl::transpose)aaaaa, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, (hipsparseOperation_t)aaaaa, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, transA, m, n, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);
  hipsparseZcsrmv(handle, transA, m, n, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);
  hipsparseDcsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);
  hipsparseZcsrmm(handle, transA, m, n, k, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = (dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y), 0)){}
  if(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = (dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y), 0);;){}
  for(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = (dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y), 0)){}
  switch(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: std::shared_ptr<dpct::sparse::optimize_info> info;
  //CHECK-NEXT: info = std::make_shared<dpct::sparse::optimize_info>();
  //CHECK-NEXT: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  //CHECK-NEXT: info.reset();
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseDcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info);
  cusparseZcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(std::shared_ptr<dpct::sparse::matrix_info> descrB) try {
//CHECK-NEXT:   /*
//CHECK-NEXT:   DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
//CHECK-NEXT:   */
//CHECK-NEXT:   /*
//CHECK-NEXT:   DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
//CHECK-NEXT:   */
//CHECK-NEXT:   return (dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y), 0);
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

