// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// RUN: dpct --format-range=none --out-root %T/cusparse-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-usm/cusparse-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
double alpha;
const double* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const double* x;
double beta;
double* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: oneapi::mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int foo(int aaaaa){
  //CHECK: oneapi::mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT: oneapi::mkl::index_base descr3 ;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;

  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_DEVICE;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_HOST);

  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: int type0 = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatDiagType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatFillMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = (oneapi::mkl::index_base)aaaaa;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatDiagType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: diag0 = (oneapi::mkl::diag)0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatFillMode was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: fill0 = (oneapi::mkl::uplo)0;
  //CHECK-NEXT: base0 = descrA;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: type0 = 0;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)aaaaa);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)aaaaa);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, dpct::get_transpose(aaaaa), alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseCreate(&handle);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)aaaaa);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseDcsrmv(handle, (hipsparseOperation_t)aaaaa, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipDoubleComplex alpha_Z, beta_Z, *csrValA_Z, *x_Z, *y_Z;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, std::complex<double>(alpha_Z.x(), alpha_Z.y()), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)y_Z);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmv(handle, transA, m, n, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, &beta_Z, y_Z);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), n, ldb, beta, y, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseDcsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), (std::complex<double>*)csrValA_Z);
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, transA, std::complex<double>(alpha_Z.x(), alpha_Z.y()), mat_handle_ct{{[0-9]+}}, (std::complex<double>*)x_Z, n, ldb, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)y_Z, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  hipsparseZcsrmm(handle, transA, m, n, k, nnz, &alpha_Z, descrA, csrValA_Z, csrRowPtrA, csrColIndA, x_Z, ldb, &beta_Z, y_Z, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: int info;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCreateSolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDcsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroySolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseDcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseZcsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseZcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_Z, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(oneapi::mkl::index_base descrB) try {
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, const_cast<int*>(csrRowPtrA), const_cast<int*>(csrColIndA), const_cast<double*>(csrValA));
//CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, const_cast<double*>(x), beta, y);
//CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseDcsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

