// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublas-create-Sgemm-destroy %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <dpct/blas_utils.hpp>
// CHECK: #include <dpct/lib_common_utils.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern dpct::queue_ptr handle2;
extern hipblasHandle_t handle2;

// CHECK: int foo2(dpct::library_data_t DT)  try {
int foo2(hipDataType DT) {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.out_of_order_queue();
  // CHECK: int status;
  // CHECK-NEXT: dpct::queue_ptr handle;
  // CHECK-NEXT: handle = &q_ct1;
  // CHECK: status = DPCT_CHECK_ERROR(handle = &q_ct1);
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  //CHECK: int Atomicsmode;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetAtomicsMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetAtomicsMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasAtomicsMode_t Atomicsmode;
  hipblasGetAtomicsMode(handle, &Atomicsmode);
  hipblasSetAtomicsMode(handle, Atomicsmode);

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::library_data_t cdt;
  //CHECK-NEXT: dpct::library_data_t cbdt;
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_HOST;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipDataType cdt;
  hipDataType cbdt;

  // CHECK: dpct::queue_ptr stream1;
  // CHECK-NEXT: stream1 = dev_ct1.create_queue();
  // CHECK-NEXT: handle = stream1;
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(handle = stream1);
  // CHECK-NEXT: stream1 = handle;
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(stream1 = handle);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N));
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_A_D)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_B_D)), N, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_C_D)), N));
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, dpct::get_transpose(trans2), oneapi::mkl::transpose::conjtrans, N, N, N, dpct::get_value(&alpha_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_A_D)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_B_D)), N, dpct::get_value(&beta_D, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_C_D)), N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N));
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
  hipblasHgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  void *alpha, *beta, *A, *B, *C;

  // CHECK: status = DPCT_CHECK_ERROR(dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_half));
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_double, N, B, dpct::library_data_t::real_double, N, beta, C, dpct::library_data_t::real_double, N, dpct::library_data_t::real_double);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, beta, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_double, N, B, dpct::library_data_t::complex_double, N, beta, C, dpct::library_data_t::complex_double, N, dpct::library_data_t::complex_double);
  status = hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_64F, N, B, HIP_R_64F, N, beta, C, HIP_R_64F, N, HIP_R_64F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_32F, N, B, HIP_C_32F, N, beta, C, HIP_C_32F, N, HIP_C_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_64F, N, B, HIP_C_64F, N, beta, C, HIP_C_64F, N, HIP_C_64F, CUBLAS_GEMM_ALGO0);

  // CHECK: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_int8, N, B, dpct::library_data_t::real_int8, N, beta, C, dpct::library_data_t::real_int32, N, dpct::library_data_t::real_int32);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_8I, N, B, HIP_R_8I, N, beta, C, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT, CUBLAS_GEMM_ALGO0);

  float2 alpha_C, beta_C;
  // CHECK: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_C, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, &beta_C, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  // CHECK: for (;;) {
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N));
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N);
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: foo(bar(DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N))));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

#define dA(i, j) *(d_A_S + (i) + (j) * N)
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(&dA(10, 20))), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, dpct::get_value(&beta_S, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N);
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, &dA(10, 20), N, d_B_S, N, &beta_S, d_C_S, N);
#undef dA(i, j)

  // CHECK: status = DPCT_CHECK_ERROR(handle = nullptr);
  // CHECK-NEXT: handle = nullptr;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}

void foo3(hipblasHandle_t handle) {
  int ver;
  // CHECK: int err = DPCT_CHECK_ERROR(dpct::mkl_get_version(dpct::version_field::major, &ver));
  int err = cublasGetVersion(handle, &ver);
}

void foo4() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  //CHECK:oneapi::mkl::blas::column_major::dgmm(*handle, oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(a_f)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_f)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(c_f)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(*handle, oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(a_d)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_d)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(c_d)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(*handle, oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(a_c)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(c_c)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(*handle, oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(a_z)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(c_z)), 2);
  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}

void foo() {
  //CHECK:const dpct::queue_ptr h_c = nullptr;
  //CHECK-NEXT:dpct::queue_ptr h = h_c;
  const hipblasHandle_t h_c = nullptr;
  hipblasHandle_t h = h_c;
}
