// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublas-create-Sgemm-destroy %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// CHECK: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <oneapi/mkl.hpp>
// CHECK: #include <dpct/lib_common_utils.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern sycl::queue* handle2;
extern hipblasHandle_t handle2;

// CHECK: int foo2(int DT)  try {
int foo2(hipDataType DT) {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.default_queue();
  // CHECK: int status;
  // CHECK-NEXT: sycl::queue* handle;
  // CHECK-NEXT: handle = &q_ct1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = &q_ct1, 0);
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  //CHECK: int Atomicsmode;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetAtomicsMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetAtomicsMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasAtomicsMode_t Atomicsmode;
  hipblasGetAtomicsMode(handle, &Atomicsmode);
  hipblasSetAtomicsMode(handle, Atomicsmode);

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: int cdt;
  //CHECK-NEXT: int cbdt;
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_HOST;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipDataType cdt;
  hipDataType cbdt;

  // CHECK: sycl::queue *stream1;
  // CHECK-NEXT: stream1 = dev_ct1.create_queue();
  // CHECK-NEXT: handle = stream1;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = stream1, 0);
  // CHECK-NEXT: stream1 = handle;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (stream1 = handle, 0);
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_D, d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, beta_D, d_C_D_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_D, d_A_D_buf_ct{{[0-9]+}}, N, d_B_D_buf_ct{{[0-9]+}}, N, beta_D, d_C_D_buf_ct{{[0-9]+}}, N);
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK: status = (oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N), 0);
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_H, d_A_H_buf_ct{{[0-9]+}}, N, d_B_H_buf_ct{{[0-9]+}}, N, beta_H, d_C_H_buf_ct{{[0-9]+}}, N);
  status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
  hipblasHgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  void *alpha, *beta, *A, *B, *C;

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_half), 0);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_double, N, B, dpct::library_data_t::real_double, N, beta, C, dpct::library_data_t::real_double, N, dpct::library_data_t::real_double);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, beta, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_double, N, B, dpct::library_data_t::complex_double, N, beta, C, dpct::library_data_t::complex_double, N, dpct::library_data_t::complex_double);
  status = hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_64F, N, B, HIP_R_64F, N, beta, C, HIP_R_64F, N, HIP_R_64F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_32F, N, B, HIP_C_32F, N, beta, C, HIP_C_32F, N, HIP_C_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_64F, N, B, HIP_C_64F, N, beta, C, HIP_C_64F, N, HIP_C_64F, CUBLAS_GEMM_ALGO0);

  // CHECK: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_int8, N, B, dpct::library_data_t::real_int8, N, beta, C, dpct::library_data_t::real_int32, N, dpct::library_data_t::real_int32);
  // CHECK-NEXT: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_8I, N, B, HIP_R_8I, N, beta, C, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT, CUBLAS_GEMM_ALGO0);

  float2 alpha_C, beta_C;
  // CHECK: {
  // CHECK-NEXT: auto A_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(A);
  // CHECK-NEXT: auto B_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(B);
  // CHECK-NEXT: auto C_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, sycl::vec<float, 1>{alpha_S}.convert<sycl::half, sycl::rounding_mode::automatic>()[0], A_buf_ct{{[0-9]+}}, N, B_buf_ct{{[0-9]+}}, N, sycl::vec<float, 1>{beta_S}.convert<sycl::half, sycl::rounding_mode::automatic>()[0], C_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(A);
  // CHECK-NEXT: auto B_buf_ct{{[0-9]+}} = dpct::get_buffer<sycl::half>(B);
  // CHECK-NEXT: auto C_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_S, A_buf_ct{{[0-9]+}}, N, B_buf_ct{{[0-9]+}}, N, beta_S, C_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(A);
  // CHECK-NEXT: auto B_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(B);
  // CHECK-NEXT: auto C_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_S, A_buf_ct{{[0-9]+}}, N, B_buf_ct{{[0-9]+}}, N, beta_S, C_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A);
  // CHECK-NEXT: auto B_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B);
  // CHECK-NEXT: auto C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), A_buf_ct{{[0-9]+}}, N, B_buf_ct{{[0-9]+}}, N, std::complex<float>(beta_C.x(), beta_C.y()), C_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: {
  // CHECK-NEXT: auto d_A_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_A_S);
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo(bar((oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, d_A_S_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N), 0)));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

#define dA(i, j) *(d_A_S + (i) + (j) * N)
  // CHECK: {
  // CHECK-NEXT: auto dA_10_20_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&dA(10, 20));
  // CHECK-NEXT: auto d_B_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_B_S);
  // CHECK-NEXT: auto d_C_S_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(d_C_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, dA_10_20_buf_ct{{[0-9]+}}, N, d_B_S_buf_ct{{[0-9]+}}, N, beta_S, d_C_S_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, &dA(10, 20), N, d_B_S, N, &beta_S, d_C_S, N);
#undef dA(i, j)

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = (handle = nullptr, 0);
  // CHECK-NEXT: handle = nullptr;
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}

void foo3(hipblasHandle_t handle) {
  int ver;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int err = (dpct::mkl_get_version(dpct::version_field::major, &ver), 0);
  int err = cublasGetVersion(handle, &ver);
}

void foo4() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  //CHECK:{
  //CHECK-NEXT:auto a_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(a_f);
  //CHECK-NEXT:auto x_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x_f);
  //CHECK-NEXT:auto c_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(c_f);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_f_buf_ct{{[0-9]+}}, 2, 0, x_f_buf_ct{{[0-9]+}}, 1, 0, c_f_buf_ct{{[0-9]+}}, 2, 0, 1);
  //CHECK-NEXT:}
  //CHECK-NEXT:{
  //CHECK-NEXT:auto a_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(a_d);
  //CHECK-NEXT:auto x_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(x_d);
  //CHECK-NEXT:auto c_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(c_d);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_d_buf_ct{{[0-9]+}}, 2, 0, x_d_buf_ct{{[0-9]+}}, 1, 0, c_d_buf_ct{{[0-9]+}}, 2, 0, 1);
  //CHECK-NEXT:}
  //CHECK-NEXT:{
  //CHECK-NEXT:auto a_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(a_c);
  //CHECK-NEXT:auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  //CHECK-NEXT:auto c_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(c_c);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_c_buf_ct{{[0-9]+}}, 2, 0, x_c_buf_ct{{[0-9]+}}, 1, 0, c_c_buf_ct{{[0-9]+}}, 2, 0, 1);
  //CHECK-NEXT:}
  //CHECK-NEXT:{
  //CHECK-NEXT:auto a_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(a_z);
  //CHECK-NEXT:auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  //CHECK-NEXT:auto c_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(c_z);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_z_buf_ct{{[0-9]+}}, 2, 0, x_z_buf_ct{{[0-9]+}}, 1, 0, c_z_buf_ct{{[0-9]+}}, 2, 0, 1);
  //CHECK-NEXT:}
  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}
