// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublas-create-Sgemm-destroy %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-create-Sgemm-destroy/cublas-create-Sgemm-destroy.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas-create-Sgemm-destroy/cublas-create-Sgemm-destroy.dp.cpp -o %T/cublas-create-Sgemm-destroy/cublas-create-Sgemm-destroy.dp.o %}
// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <dpct/blas_utils.hpp>
// CHECK: #include <dpct/lib_common_utils.hpp>
#include <cstdio>
#include "hipblas.h"
#include <hip/hip_runtime.h>

void foo (hipblasStatus_t s){
}
hipblasStatus_t bar (hipblasStatus_t s){
  return s;
}

// CHECK: extern dpct::blas::descriptor_ptr handle2;
extern hipblasHandle_t handle2;

// CHECK: int foo2(dpct::library_data_t DT)  try {
int foo2(hipDataType DT) {
  // CHECK: int status;
  // CHECK-NEXT: dpct::blas::descriptor_ptr handle;
  // CHECK-NEXT: handle = new dpct::blas::descriptor();
  // CHECK: status = DPCT_CHECK_ERROR(handle = new dpct::blas::descriptor());
  // CHECK-NEXT: if (status != 0) {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasCreate(&handle);
  status = hipblasCreate(&handle);
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  //CHECK: int Atomicsmode;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetAtomicsMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetAtomicsMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasAtomicsMode_t Atomicsmode;
  hipblasGetAtomicsMode(handle, &Atomicsmode);
  hipblasSetAtomicsMode(handle, Atomicsmode);

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::library_data_t cdt;
  //CHECK-NEXT: dpct::library_data_t cbdt;
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_HOST;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_DEVICE);
  hipDataType cdt;
  hipDataType cbdt;

  // CHECK: dpct::queue_ptr stream1;
  // CHECK-NEXT: stream1 = dpct::get_current_device().create_queue();
  // CHECK-NEXT: handle->set_queue(stream1);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(handle->set_queue(stream1));
  // CHECK-NEXT: stream1 = &(handle->get_queue());
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(stream1 = &(handle->get_queue()));
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  hipblasSetStream(handle, stream1);
  status = hipblasSetStream(handle, stream1);
  hipblasGetStream(handle, &stream1);
  status = hipblasGetStream(handle, &stream1);


  int N = 275;
  float *d_A_S = 0;
  float *d_B_S = 0;
  float *d_C_S = 0;
  float alpha_S = 1.0f;
  float beta_S = 0.0f;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false)));
  // CHECK: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false));
  status = hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  double *d_A_D = 0;
  double *d_B_D = 0;
  double *d_C_D = 0;
  double alpha_D = 1.0;
  double beta_D = 0.0;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_D, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_A_D)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_B_D)), N, beta_D, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_C_D)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false)));
  // CHECK: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans2), oneapi::mkl::transpose::conjtrans, N, N, N, alpha_D, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_A_D)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_B_D)), N, beta_D, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d_C_D)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false));
  status = hipblasDgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_A_H)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_B_H)), N, beta_H, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_C_H)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false)));
  // CHECK: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans2), oneapi::mkl::transpose::conjtrans, N, N, N, alpha_H, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_A_H)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_B_H)), N, beta_H, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(d_C_H)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false));
  status = hipblasHgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);
  hipblasHgemm(handle, (hipblasOperation_t)trans2, (hipblasOperation_t)2, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  void *alpha, *beta, *A, *B, *C;

  // CHECK: status = DPCT_CHECK_ERROR(dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_half));
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_double, N, B, dpct::library_data_t::real_double, N, beta, C, dpct::library_data_t::real_double, N, dpct::library_data_t::real_double);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, beta, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::complex_double, N, B, dpct::library_data_t::complex_double, N, beta, C, dpct::library_data_t::complex_double, N, dpct::library_data_t::complex_double);
  status = hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_16F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_16F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_16F, N, B, HIP_R_16F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_64F, N, B, HIP_R_64F, N, beta, C, HIP_R_64F, N, HIP_R_64F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_32F, N, B, HIP_C_32F, N, beta, C, HIP_C_32F, N, HIP_C_32F, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_C_64F, N, B, HIP_C_64F, N, beta, C, HIP_C_64F, N, HIP_C_64F, CUBLAS_GEMM_ALGO0);

  // CHECK: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_int8, N, B, dpct::library_data_t::real_int8, N, beta, C, dpct::library_data_t::real_int32, N, dpct::library_data_t::real_int32);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_8I, N, B, HIP_R_8I, N, beta, C, HIP_R_32I, N, HIP_R_32I, CUBLAS_GEMM_ALGO0);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, DT, N, B, DT, N, beta, C, DT, N, DT, CUBLAS_GEMM_ALGO0);

  float2 alpha_C, beta_C;
  // CHECK: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  // CHECK-NEXT: dpct::blas::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_C, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, &beta_C, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  // CHECK: for (;;) {
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false)));
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    status = hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;

  // CHECK: for (;;) {
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::trans, oneapi::mkl::transpose::trans, N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false));
  // CHECK-NEXT: beta_S = beta_S + 1;
  // CHECK-NEXT: }
  // CHECK-NEXT: alpha_S = alpha_S + 1;
  for (;;) {
    hipblasSgemm(handle, HIPBLAS_OP_T, HIPBLAS_OP_T, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
    beta_S = beta_S + 1;
  }
  alpha_S = alpha_S + 1;


  // CHECK: foo(bar(DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_A_S)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false)))));
  foo(bar(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)));

#define dA(i, j) *(d_A_S + (i) + (j) * N)
  // CHECK: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(&dA(10, 20))), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_B_S)), N, beta_S, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(d_C_S)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), false));
  hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_S, &dA(10, 20), N, d_B_S, N, &beta_S, d_C_S, N);
#undef dA(i, j)

  // CHECK: status = DPCT_CHECK_ERROR(delete (handle));
  // CHECK-NEXT: delete (handle);
  // CHECK-NEXT: return 0;
  status = hipblasDestroy(handle);
  hipblasDestroy(handle);
  return 0;
}

void foo3(hipblasHandle_t handle) {
  int ver;
  // CHECK: int err = DPCT_CHECK_ERROR(dpct::mkl_get_version(dpct::version_field::major, &ver));
  int err = cublasGetVersion(handle, &ver);
}

void foo4() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  //CHECK:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(a_f)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_f)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(c_f)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(a_d)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_d)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(c_d)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(a_c)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(c_c)), 2);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, 2, 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(a_z)), 2, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), 1, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(c_z)), 2);
  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}

void foo() {
  //CHECK:const dpct::blas::descriptor_ptr h_c = nullptr;
  //CHECK-NEXT:dpct::blas::descriptor_ptr h = h_c;
  const hipblasHandle_t h_c = nullptr;
  hipblasHandle_t h = h_c;
}

#define CHECK_INTERNAL(err)                                                    \
  { auto err_ = (err); }

#define CHECK(err) CHECK_INTERNAL(err)

void foo2(hipblasHandle_t *handles, int i, hipStream_t s) {
  // CHECK: CHECK(DPCT_CHECK_ERROR(handles[i] = new dpct::blas::descriptor()));
  // CHECK-NEXT: CHECK(DPCT_CHECK_ERROR(delete (handles[i])));
  // CHECK-NEXT: CHECK(DPCT_CHECK_ERROR(handles[i]->set_queue(s)));
  // CHECK-NEXT: CHECK(DPCT_CHECK_ERROR(s = &(handles[i]->get_queue())));
  CHECK(hipblasCreate(&handles[i]));
  CHECK(hipblasDestroy(handles[i]));
  CHECK(hipblasSetStream(handles[i], s));
  CHECK(hipblasGetStream(handles[i], &s));
}
