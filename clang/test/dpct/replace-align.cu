// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7
// RUN: dpct --format-range=none -out-root %T/replace-align %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/replace-align/replace-align.dp.cpp


#include <hip/hip_runtime.h>
#include <hip/hip_fp8.h>
// CHECK:typedef struct __dpct_align__(4) dpct_type_{{[a-f0-9]+}}
typedef struct __align__(4)
{
    unsigned char r, g, b, a;
}
T0;

// CHECK:class __dpct_align__(8) T1 {
class __align__(8) T1 {
    unsigned int l, a;
};

// CHECK:struct __attribute__((aligned(16))) T2
struct __attribute__((aligned(16))) T2
{
    unsigned int r, g, b;
};

// CHECK:class __dpct_align__(16) color {
class __CUDA_ALIGN__(16) color {
    unsigned int r, g, b;
};
