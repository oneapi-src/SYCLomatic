// RUN: dpct --format-range=none --usm-level=none -out-root %T/texture_object_driver %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/texture_object_driver/texture_object_driver.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/texture_object_driver/texture_object_driver.dp.cpp -o %T/texture_object_driver/texture_object_driver.dp.o %}

#include "hip/hip_runtime.h"
#include <stdio.h>

#define cudaCheck(stmt) do {                         \
  int err = stmt;                            \
  if (err != hipSuccess) {                          \
    char msg[256];                                   \
    sprintf(msg, "%s in file %s, function %s, line %d\n", #stmt,__FILE__,__FUNCTION__,__LINE__); \
  }                                                  \
} while(0)

void func(int i) {}

template <typename T>
void funcT(T t) {}

// CHECK: template <class T> void BindTextureObject(dpct::image_matrix_p &data, dpct::image_wrapper_base_p &tex) {
// CHECK-NEXT:   dpct::image_data res42;
// CHECK-NEXT:   dpct::sampling_info texDesc42;
// CHECK-NEXT:   res42.set_data_type(dpct::image_data_type::matrix);
// CHECK-NEXT:   res42.set_data_ptr(data);
// CHECK-NEXT:   texDesc42.set(sycl::addressing_mode::clamp_to_edge);
// CHECK-NEXT:   texDesc42.set(sycl::filtering_mode::nearest);
// CHECK-NEXT:   data = (dpct::image_matrix_p)res42.get_data_ptr();
// CHECK-NEXT:   tex = dpct::create_image_wrapper(res42, texDesc42);
// CHECK-NEXT: }
template <class T> void BindTextureObject(hipArray_t &data, hipTextureObject_t &tex) {
  HIP_RESOURCE_DESC res42;
  HIP_TEXTURE_DESC texDesc42;
  res42.resType = HIP_RESOURCE_TYPE_ARRAY;
  res42.res.array.hArray = data;
  texDesc42.addressMode[0] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.addressMode[1] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.addressMode[2] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.filterMode = HIP_TR_FILTER_MODE_POINT;
  data = res42.res.array.hArray;
  hipTexObjectCreate(&tex, &res42, &texDesc42, NULL);
}

int main() {

  // CHECK: sycl::float4 *d_data42;
  // CHECK-NEXT: dpct::image_matrix_p a42;
  // CHECK-NEXT: dpct::image_desc desc42;
  // CHECK-NEXT: desc42.channel_num = 4;
  // CHECK-NEXT: desc42.channel_type = sycl::image_channel_type::fp32;
  // CHECK-NEXT: desc42.width = 32;
  // CHECK-NEXT: desc42.height = 32;
  // CHECK-NEXT: a42 = new dpct::image_matrix((&desc42)->channel_type, (&desc42)->channel_num, (&desc42)->width, (&desc42)->height);
  // CHECK-NEXT: dpct::image_wrapper_base_p tex42;
  // CHECK-NEXT: dpct::image_data res42;
  // CHECK-NEXT: dpct::sampling_info texDesc42;
  // CHECK-NEXT: res42.set_data_type(dpct::image_data_type::pitch);
  // CHECK-NEXT: res42.set_data_ptr((dpct::device_ptr)d_data42);
  // CHECK-NEXT: res42.set_x(sizeof(sycl::float4) * 32);
  // CHECK-NEXT: res42.set_y(32);
  // CHECK-NEXT: res42.set_pitch(sizeof(sycl::float4) * 32);
  // CHECK-NEXT: res42.set_channel_num(4);
  // CHECK-NEXT: res42.set_channel_type(sycl::image_channel_type::fp32);
  // CHECK-NEXT: res42.set_data_type(dpct::image_data_type::matrix);
  // CHECK-NEXT: res42.set_data_ptr(a42);
  // CHECK-NEXT: texDesc42.set(sycl::addressing_mode::clamp_to_edge, sycl::filtering_mode::nearest, sycl::coordinate_normalization_mode::normalized);
  // CHECK-NEXT: tex42 = dpct::create_image_wrapper(res42, texDesc42);
  float4 *d_data42;
  hipArray_t a42;
  HIP_ARRAY_DESCRIPTOR desc42;
  desc42.NumChannels = 4;
  desc42.Format = HIP_AD_FORMAT_FLOAT;
  desc42.Width = 32;
  desc42.Height = 32;
  hipArrayCreate(&a42, &desc42);
  hipTextureObject_t tex42;
  HIP_RESOURCE_DESC res42;
  HIP_TEXTURE_DESC texDesc42;
  res42.resType = HIP_RESOURCE_TYPE_PITCH2D;
  res42.res.pitch2D.devPtr = (hipDeviceptr_t)d_data42;
  res42.res.pitch2D.width = sizeof(float4) * 32;
  res42.res.pitch2D.height = 32;
  res42.res.pitch2D.pitchInBytes = sizeof(float4) * 32;
  res42.res.pitch2D.numChannels = 4;
  res42.res.pitch2D.format = HIP_AD_FORMAT_FLOAT;
  res42.resType = HIP_RESOURCE_TYPE_ARRAY;
  res42.res.array.hArray = a42;
  texDesc42.addressMode[0] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.addressMode[1] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.addressMode[2] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc42.filterMode = HIP_TR_FILTER_MODE_POINT;
  texDesc42.flags = HIP_TRSF_READ_AS_INTEGER | HIP_TRSF_NORMALIZED_COORDINATES;
  hipTexObjectCreate(&tex42, &res42, &texDesc42, NULL);

  // CHECK: sycl::uint2 *d_data21;
  // CHECK-NEXT: d_data21 = (sycl::uint2 *)dpct::dpct_malloc(sizeof(sycl::uint2) * 32);
  // CHECK-NEXT: dpct::image_wrapper_base_p tex21;
  // CHECK-NEXT: dpct::image_data res21;
  // CHECK-NEXT: dpct::sampling_info texDesc21;
  // CHECK-NEXT: res21.set_data_type(dpct::image_data_type::linear);
  // CHECK-NEXT: res21.set_data_ptr((dpct::device_ptr)d_data21);
  // CHECK-NEXT: res21.set_channel_num(2);
  // CHECK-NEXT: res21.set_channel_type(sycl::image_channel_type::unsigned_int32);
  // CHECK-NEXT: res21.set_x(32*sizeof(sycl::uint2));
  // CHECK-NEXT: unsigned chnX = res21.get_channel_num();
  // CHECK-NEXT: sycl::image_channel_type formatKind = res21.get_channel_type();
  // CHECK-NEXT: texDesc21.set(sycl::addressing_mode::clamp_to_edge, sycl::filtering_mode::linear, sycl::coordinate_normalization_mode::normalized);
  // CHECK-NEXT: tex21 = dpct::create_image_wrapper(res21, texDesc21);
  uint2 *d_data21;
  hipMalloc(&d_data21, sizeof(uint2) * 32);
  hipTextureObject_t tex21;
  HIP_RESOURCE_DESC res21;
  HIP_TEXTURE_DESC texDesc21;
  res21.resType = HIP_RESOURCE_TYPE_LINEAR;
  res21.res.linear.devPtr = (hipDeviceptr_t)d_data21;
  res21.res.linear.numChannels = 2;
  res21.res.linear.format = HIP_AD_FORMAT_UNSIGNED_INT32;
  res21.res.linear.sizeInBytes = 32*sizeof(uint2);
  unsigned chnX = res21.res.linear.numChannels;
  hipArray_Format formatKind = res21.res.linear.format;
  texDesc21.addressMode[0] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc21.addressMode[1] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc21.addressMode[2] = HIP_TR_ADDRESS_MODE_CLAMP;
  texDesc21.filterMode = HIP_TR_FILTER_MODE_LINEAR;
  texDesc21.flags = HIP_TRSF_NORMALIZED_COORDINATES | HIP_TRSF_READ_AS_INTEGER;
  hipTexObjectCreate(&tex21, &res21, &texDesc21, NULL);
  
  // CHECK: texDesc42 = tex42->get_sampling_info();
  // CHECK-NEXT: res42 = tex42->get_data();
  hipTexObjectGetTextureDesc(&texDesc42, tex42);
  hipTexObjectGetResourceDesc(&res42, tex42);

  // CHECK: delete tex42;
  // CHECK-NEXT: delete tex21;
  hipTexObjectDestroy(tex42);
  hipTexObjectDestroy(tex21);

  // CHECK: delete a42;
  hipArrayDestroy(a42);

  // CHECK: dpct::dpct_free(d_data42);
  // CHECK-NEXT: dpct::dpct_free(d_data21);
  hipFree(d_data42);
  hipFree(d_data21);

  // Test IsAssigned
  {
    int errorCode;
    // CHECK: errorCode = DPCT_CHECK_ERROR(tex21 = dpct::create_image_wrapper(res21, texDesc21));
    errorCode = hipTexObjectCreate(&tex21, &res21, &texDesc21, NULL);
    // CHECK: cudaCheck(DPCT_CHECK_ERROR(tex21 = dpct::create_image_wrapper(res21, texDesc21)));
    cudaCheck(hipTexObjectCreate(&tex21, &res21, &texDesc21, NULL));
    // CHECK: func(DPCT_CHECK_ERROR(tex21 = dpct::create_image_wrapper(res21, texDesc21)));
    func(hipTexObjectCreate(&tex21, &res21, &texDesc21, NULL));
    // CHECK: funcT(DPCT_CHECK_ERROR(tex21 = dpct::create_image_wrapper(res21, texDesc21)));
    funcT(hipTexObjectCreate(&tex21, &res21, &texDesc21, NULL));

    // CHECK: errorCode = DPCT_CHECK_ERROR(delete tex21);
    errorCode = hipTexObjectDestroy(tex21);
    // CHECK: cudaCheck(DPCT_CHECK_ERROR(delete tex21));
    cudaCheck(hipTexObjectDestroy(tex21));
    // CHECK: func(DPCT_CHECK_ERROR(delete tex21));
    func(hipTexObjectDestroy(tex21));
    // CHECK: funcT(DPCT_CHECK_ERROR(delete tex21));
    funcT(hipTexObjectDestroy(tex21));
  }
}

void foo(){
  hipArray_t a42;
  HIP_RESOURCE_DESC res42;
  // CHECK: res42.set_data_type(dpct::image_data_type::matrix);
  // CHECK-NEXT: res42.set_data_ptr(a42);
  res42.resType = HIP_RESOURCE_TYPE_ARRAY;
  res42.res.array.hArray = a42;

  float4 *d_data42;
  // CHECK: res42.set_data_type(dpct::image_data_type::pitch);
  // CHECK-NEXT: res42.set_data_ptr((dpct::device_ptr)d_data42);
  // CHECK-NEXT: res42.set_channel_num(4);
  // CHECK-NEXT: res42.set_channel_type(sycl::image_channel_type::fp32);
  // CHECK-NEXT: res42.set_x(sizeof(sycl::float4) * 32);
  // CHECK-NEXT: res42.set_y(32);
  // CHECK-NEXT: res42.set_pitch(sizeof(sycl::float4) * 32);
  res42.resType = HIP_RESOURCE_TYPE_PITCH2D;
  res42.res.pitch2D.devPtr = (hipDeviceptr_t)d_data42;
  res42.res.pitch2D.numChannels = 4;
  res42.res.pitch2D.format = HIP_AD_FORMAT_FLOAT;
  res42.res.pitch2D.width = sizeof(float4) * 32;
  res42.res.pitch2D.height = 32;
  res42.res.pitch2D.pitchInBytes = sizeof(float4) * 32;

  uint2 *d_data21;
  // CHECK: res42.set_data_type(dpct::image_data_type::linear);
  // CHECK-NEXT: res42.set_data_ptr((dpct::device_ptr)d_data21);
  // CHECK-NEXT: res42.set_x(sizeof(sycl::float4) * 32);
  // CHECK-NEXT: res42.set_channel_num(4);
  // CHECK-NEXT: res42.set_channel_type(sycl::image_channel_type::fp32);
  res42.resType = HIP_RESOURCE_TYPE_LINEAR;
  res42.res.linear.devPtr = (hipDeviceptr_t)d_data21;
  res42.res.linear.sizeInBytes = sizeof(float4) * 32;
  res42.res.pitch2D.numChannels = 4;
  res42.res.pitch2D.format = HIP_AD_FORMAT_FLOAT;

  // CHECK:  dpct::sampling_info tex_tmp;
  // CHECK-NEXT:   tex_tmp.set(sycl::addressing_mode::clamp_to_edge, sycl::filtering_mode::nearest, sycl::coordinate_normalization_mode::unnormalized);
  // CHECK-NEXT:   sycl::addressing_mode addr = tex_tmp.get_addressing_mode();
  // CHECK-NEXT:   sycl::filtering_mode filter = tex_tmp.get_filtering_mode();
  // CHECK-NEXT:   int flags = tex_tmp.is_coordinate_normalized();
  HIP_TEXTURE_DESC tex_tmp;
  tex_tmp.flags = HIP_TRSF_READ_AS_INTEGER;
  tex_tmp.addressMode[0] = HIP_TR_ADDRESS_MODE_CLAMP;
  tex_tmp.filterMode = HIP_TR_FILTER_MODE_POINT;
  HIPaddress_mode_enum addr = tex_tmp.addressMode[0];
  HIPfilter_mode_enum filter = tex_tmp.filterMode;
  int flags = tex_tmp.flags;
  
  // CHECK: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: tex_tmp.set_coordinate_normalization_mode(flags & 0x02);
  // CHECK-NEXT: tex_tmp.set(sycl::addressing_mode::clamp_to_edge);
  // CHECK-NEXT: if (true) {
  // CHECK-NEXT:   tex_tmp.set(sycl::addressing_mode::clamp_to_edge, sycl::filtering_mode::nearest, sycl::coordinate_normalization_mode::normalized);
  // CHECK-NEXT: }
  // CHECK-NEXT: tex_tmp.set(sycl::filtering_mode::linear);
  // CHECK-NEXT: addr = tex_tmp.get_addressing_mode();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1074:{{[0-9]+}}: The SYCL Image class does not support some of the flags used in the original code. Unsupported flags were ignored. Data read from SYCL Image could not be normalized as specified in the original code.
  // CHECK-NEXT: */
  // CHECK-NEXT: tex_tmp.set(sycl::addressing_mode::clamp_to_edge, sycl::filtering_mode::nearest, sycl::coordinate_normalization_mode::normalized);
  tex_tmp.flags = flags;
  tex_tmp.addressMode[0] = HIP_TR_ADDRESS_MODE_CLAMP;
  if (true) {
    tex_tmp.filterMode = HIP_TR_FILTER_MODE_POINT;
    tex_tmp.addressMode[1] = HIP_TR_ADDRESS_MODE_CLAMP;
    tex_tmp.addressMode[2] = HIP_TR_ADDRESS_MODE_CLAMP;
    tex_tmp.flags = 3;
  }
  tex_tmp.filterMode = HIP_TR_FILTER_MODE_LINEAR;
  addr = tex_tmp.addressMode[0];
  tex_tmp.filterMode = HIP_TR_FILTER_MODE_POINT;
  tex_tmp.addressMode[2] = HIP_TR_ADDRESS_MODE_CLAMP;
  tex_tmp.flags = HIP_TRSF_NORMALIZED_COORDINATES;
}

