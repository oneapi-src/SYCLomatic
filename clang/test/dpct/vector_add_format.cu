// RUN: mkdir %T/vector_add_format
// RUN: cat %s > %T/vector_add_format/vector_add_format.cu
// RUN: cd %T/vector_add_format
// RUN: dpct -out-root %T/vector_add_format vector_add_format.cu --cuda-include-path="%cuda-path/include" -- -std=c++14  -x cuda --cuda-host-only
// RUN: FileCheck -strict-whitespace %s --match-full-lines --input-file %T/vector_add_format/vector_add_format.dp.cpp
// RUN: cd ..
// RUN: rm -rf ./vector_add_format

#include <hip/hip_runtime.h>
#include <stdio.h>
#define VECTOR_SIZE 256

//CHECK:__dpct_inline__ void VectorAddKernel(float *A, float *B, float *C,
// CHECK-NEXT:                                     const sycl::nd_item<3> &item_ct1)
//CHECK-NEXT:{
//CHECK-NEXT:    A[item_ct1.get_local_id(2)] = item_ct1.get_local_id(2) + 1.0f;
//CHECK-NEXT:    B[item_ct1.get_local_id(2)] = item_ct1.get_local_id(2) + 1.0f;
//CHECK-NEXT:    C[item_ct1.get_local_id(2)] =
//CHECK-NEXT:        A[item_ct1.get_local_id(2)] + B[item_ct1.get_local_id(2)];
//CHECK-NEXT:}
__global__ void VectorAddKernel(float* A, float* B, float* C)
{
    A[threadIdx.x] = threadIdx.x + 1.0f;
    B[threadIdx.x] = threadIdx.x + 1.0f;
    C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}



int main()
{
  //      CHECK:    sycl::device dev_ct1;
  // CHECK-NEXT:    sycl::queue q_ct1(dev_ct1,
  // CHECK-NEXT:                      sycl::property_list{sycl::property::queue::in_order()});
  float *d_A, *d_B, *d_C;

  //     CHECK:  d_A = sycl::malloc_device<float>(VECTOR_SIZE, q_ct1);
  // CHECK-NEXT:  d_B = sycl::malloc_device<float>(VECTOR_SIZE, q_ct1);
  // CHECK-NEXT:  d_C = sycl::malloc_device<float>(VECTOR_SIZE, q_ct1);
  hipMalloc(&d_A, VECTOR_SIZE * sizeof(float));
  hipMalloc(&d_B, VECTOR_SIZE * sizeof(float));
  hipMalloc(&d_C, VECTOR_SIZE * sizeof(float));

  //     CHECK:  q_ct1.parallel_for(sycl::nd_range<3>(sycl::range<3>(1, 1, VECTOR_SIZE),
  // CHECK-NEXT:                                       sycl::range<3>(1, 1, VECTOR_SIZE)),
  // CHECK-NEXT:                     [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:                       VectorAddKernel(d_A, d_B, d_C, item_ct1);
  // CHECK-NEXT:                     });
  VectorAddKernel<<<1, VECTOR_SIZE>>>(d_A, d_B, d_C);

  //     CHECK:  float Result[VECTOR_SIZE] = {};
  // CHECK-NEXT:  q_ct1.memcpy(Result, d_C, VECTOR_SIZE * sizeof(float)).wait();
  float Result[VECTOR_SIZE] = {};
  hipMemcpy(Result, d_C, VECTOR_SIZE * sizeof(float), hipMemcpyDeviceToHost);

  //     CHECK:  sycl::free(d_A, q_ct1);
  // CHECK-NEXT:  sycl::free(d_B, q_ct1);
  // CHECK-NEXT:  sycl::free(d_C, q_ct1);
  hipFree(d_A);
  hipFree(d_B);
  hipFree(d_C);

  for (int i = 0; i < VECTOR_SIZE; i++) {
    if (i % 16 == 0) {
      printf("\n");
    }
    printf("%f ", Result[i]);
  }

    return 0;
}
