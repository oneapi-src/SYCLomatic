// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0
// RUN: dpct --format-range=none --out-root %T/cusparse-after101 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cusparse-after101/cusparse-after101.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include <hipsparse.h>

void foo0() {
  // CHECK: dpct::sparse::conversion_scope a = dpct::sparse::conversion_scope::index;
  // CHECK-NEXT: dpct::sparse::conversion_scope b = dpct::sparse::conversion_scope::index_and_value;
  hipsparseAction_t a = HIPSPARSE_ACTION_SYMBOLIC;
  hipsparseAction_t b = HIPSPARSE_ACTION_NUMERIC;
}

void foo1() {
  hipsparseHandle_t handle;
  float* a_val;
  int* a_row_ptr;
  int* a_col_ind;
  float* b_val;
  int* b_col_ptr;
  int* b_row_ind;

  size_t ws_size = 0;
  void *ws;

  // CHECK: ws_size = 0;
  // CHECK-NEXT: dpct::sparse::csr2csc(handle->get_queue(), 3, 4, 7, a_val, a_row_ptr, a_col_ind, b_val, b_col_ptr, b_row_ind, dpct::library_data_t::real_float, dpct::sparse::conversion_scope::index_and_value, oneapi::mkl::index_base::zero);
  hipsparseCsr2cscEx2_bufferSize(handle, 3, 4, 7, a_val, a_row_ptr, a_col_ind, b_val, b_col_ptr, b_row_ind, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, &ws_size);
  hipsparseCsr2cscEx2(handle, 3, 4, 7, a_val, a_row_ptr, a_col_ind, b_val, b_col_ptr, b_row_ind, HIP_R_32F, HIPSPARSE_ACTION_NUMERIC, HIPSPARSE_INDEX_BASE_ZERO, HIPSPARSE_CSR2CSC_ALG1, ws);
}
