// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasRegularCZ %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasRegularCZ/cublasRegularCZ.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hipblas.h>

int foo();

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::iamax(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one));
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one);
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::iamax(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one));
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one);
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::iamin(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one));
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one);
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::iamin(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one));
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_buf_ct{{[0-9]+}} = sycl::buffer<int>(sycl::range<1>(1));
  // CHECK-NEXT: sycl::buffer<int64_t> res_temp_buf_ct{{[0-9]+}}(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result)) {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(result);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_buf_ct{{[0-9]+}} = sycl::buffer<int>(result, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, res_temp_buf_ct{{[0-9]+}}, oneapi::mkl::index_base::one);
  // CHECK-NEXT: result_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::write>()[0] = (int)res_temp_buf_ct{{[0-9]+}}.get_access<sycl::access_mode::read>()[0];
  // CHECK-NEXT: }
  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_f)) {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::asum(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, result_f_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_f)) {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, result_f_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_d)) {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::asum(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, result_d_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_d)) {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, result_d_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(*handle, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(*handle, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(*handle, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(*handle, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_c)) {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)result_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dotu(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_c)) {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)result_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_c)) {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)result_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dotc(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_c)) {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)result_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_z)) {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)result_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dotu(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_z)) {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)result_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_z)) {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)result_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dotc(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_z)) {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)result_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_f)) {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::nrm2(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, result_f_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_f)) {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(result_f);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(result_f, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, result_f_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_d)) {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::nrm2(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, result_d_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(result_d)) {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(result_d);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   result_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(result_d, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, result_d_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, dpct::get_value(c_f, *handle), dpct::get_value(s_f, *handle)));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, dpct::get_value(c_f, *handle), dpct::get_value(s_f, *handle));
  // CHECK-NEXT: }
  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, dpct::get_value(c_d, *handle), dpct::get_value(s_d, *handle)));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, dpct::get_value(c_d, *handle), dpct::get_value(s_d, *handle));
  // CHECK-NEXT: }
  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::rot(*handle, n, x_c, dpct::library_data_t::complex_float, incx, y_c, dpct::library_data_t::complex_float, incy, c_f, s_c, dpct::library_data_t::complex_float));
  // CHECK-NEXT: dpct::rot(*handle, n, x_c, dpct::library_data_t::complex_float, incx, y_c, dpct::library_data_t::complex_float, incy, c_f, s_c, dpct::library_data_t::complex_float);
  status = hipblasCrot(handle, n, x_c, incx, y_c, incy, c_f, s_c);
  hipblasCrot(handle, n, x_c, incx, y_c, incy, c_f, s_c);

  // CHECK: status = DPCT_CHECK_ERROR(dpct::rot(*handle, n, x_z, dpct::library_data_t::complex_double, incx, y_z, dpct::library_data_t::complex_double, incy, c_d, s_z, dpct::library_data_t::complex_double));
  // CHECK-NEXT: dpct::rot(*handle, n, x_z, dpct::library_data_t::complex_double, incx, y_z, dpct::library_data_t::complex_double, incy, c_d, s_z, dpct::library_data_t::complex_double);
  status = hipblasZrot(handle, n, x_z, incx, y_z, incy, c_d, s_z);
  hipblasZrot(handle, n, x_z, incx, y_z, incy, c_d, s_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: auto c_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto s_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_c)) {
  // CHECK-NEXT:   x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT:   y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT:   c_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(c_f);
  // CHECK-NEXT:   s_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(s_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)x_c, sycl::range<1>(1));
  // CHECK-NEXT:   y_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)y_c, sycl::range<1>(1));
  // CHECK-NEXT:   c_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(c_f, sycl::range<1>(1));
  // CHECK-NEXT:   s_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)s_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(*handle, x_c_buf_ct{{[0-9]+}}, y_c_buf_ct{{[0-9]+}}, c_f_buf_ct{{[0-9]+}}, s_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: auto c_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(sycl::range<1>(1));
  // CHECK-NEXT: auto s_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_c)) {
  // CHECK-NEXT:   x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT:   y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT:   c_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(c_f);
  // CHECK-NEXT:   s_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(s_c);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)x_c, sycl::range<1>(1));
  // CHECK-NEXT:   y_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)y_c, sycl::range<1>(1));
  // CHECK-NEXT:   c_f_buf_ct{{[0-9]+}} = sycl::buffer<float>(c_f, sycl::range<1>(1));
  // CHECK-NEXT:   s_c_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<float>>((std::complex<float>*)s_c, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, x_c_buf_ct{{[0-9]+}}, y_c_buf_ct{{[0-9]+}}, c_f_buf_ct{{[0-9]+}}, s_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: auto c_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto s_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_z)) {
  // CHECK-NEXT:   x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT:   y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT:   c_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(c_d);
  // CHECK-NEXT:   s_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(s_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)x_z, sycl::range<1>(1));
  // CHECK-NEXT:   y_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)y_z, sycl::range<1>(1));
  // CHECK-NEXT:   c_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(c_d, sycl::range<1>(1));
  // CHECK-NEXT:   s_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)s_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(*handle, x_z_buf_ct{{[0-9]+}}, y_z_buf_ct{{[0-9]+}}, c_d_buf_ct{{[0-9]+}}, s_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: auto c_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(sycl::range<1>(1));
  // CHECK-NEXT: auto s_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>(sycl::range<1>(1));
  // CHECK-NEXT: if (dpct::is_device_ptr(x_z)) {
  // CHECK-NEXT:   x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT:   y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT:   c_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(c_d);
  // CHECK-NEXT:   s_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(s_z);
  // CHECK-NEXT: } else {
  // CHECK-NEXT:   x_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)x_z, sycl::range<1>(1));
  // CHECK-NEXT:   y_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)y_z, sycl::range<1>(1));
  // CHECK-NEXT:   c_d_buf_ct{{[0-9]+}} = sycl::buffer<double>(c_d, sycl::range<1>(1));
  // CHECK-NEXT:   s_z_buf_ct{{[0-9]+}} = sycl::buffer<std::complex<double>>((std::complex<double>*)s_z, sycl::range<1>(1));
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, x_z_buf_ct{{[0-9]+}}, y_z_buf_ct{{[0-9]+}}, c_d_buf_ct{{[0-9]+}}, s_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(*handle, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(*handle, n, x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(*handle, n, x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, m, n, kl, ku, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(*handle, oneapi::mkl::transpose::nontrans, m, n, kl, ku, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgbmv(handle, (hipblasOperation_t)trans0, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(*handle, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, m, n, kl, ku, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(*handle, oneapi::mkl::transpose::nontrans, m, n, kl, ku, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgbmv(handle, (hipblasOperation_t)trans1, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCgemv(handle, (hipblasOperation_t)trans2, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZgemv(handle, (hipblasOperation_t)0, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(*handle, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::geru(*handle, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gerc(*handle, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gerc(*handle, m, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(*handle, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::geru(*handle, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gerc(*handle, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gerc(*handle, m, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  int fill0 = 0;
  int fill1 = 1;
  int diag0 = 0;
  int diag1 = 1;
  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, (oneapi::mkl::diag)diag0, n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(*handle, fill1==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, (oneapi::mkl::diag)diag1, n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbmv(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::lower, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::nonunit,  n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit,  n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbsv(handle, (hipblasFillMode_t)0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::unit,  n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit,  n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbsv(handle, (hipblasFillMode_t)1, (hipblasOperation_t)trans0, (hipblasDiagType_t)1,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_c_buf_ct{{[0-9]+}}, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_c_buf_ct{{[0-9]+}}, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_z_buf_ct{{[0-9]+}}, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_z_buf_ct{{[0-9]+}}, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_c_buf_ct{{[0-9]+}}, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_c_buf_ct{{[0-9]+}}, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_z_buf_ct{{[0-9]+}}, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_z_buf_ct{{[0-9]+}}, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, n, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemv(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChemv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemv(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhemv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, k, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hbmv(*handle, oneapi::mkl::uplo::lower, n, k, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hbmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, k, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hbmv(*handle, oneapi::mkl::uplo::lower, n, k, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpmv(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChpmv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpmv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpmv(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhpmv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_f, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_d, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr2(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr2(*handle, oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, y_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, lda, y_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCsymv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, y_c, incx, beta_c, result_c, incy);
  hipblasCsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, lda, y_c, incx, beta_c, result_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, y_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, lda, y_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZsymv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, y_z, incx, beta_z, result_z, incy);
  hipblasZsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, lda, y_z, incx, beta_z, result_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCsyr(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, result_c, lda);
  hipblasCsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, incx, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZsyr(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, result_z, lda);
  hipblasZsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, incx, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCsyr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(*handle, oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZsyr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);


  // level 3
  int N = 100;
  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, N, N, N, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, N, y_c_buf_ct{{[0-9]+}}, N, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, N));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, N, y_c_buf_ct{{[0-9]+}}, N, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  status = hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, N, N, N, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, N, y_z_buf_ct{{[0-9]+}}, N, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, N));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, N, y_z_buf_ct{{[0-9]+}}, N, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, N);
  // CHECK-NEXT: }
  status = hipblasZgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::trans, N, N, N, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, N, 16, y_c_buf_ct{{[0-9]+}}, N, 16, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, N, 16, 10));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, dpct::get_value(alpha_c, *handle), x_c_buf_ct{{[0-9]+}}, N, 16, y_c_buf_ct{{[0-9]+}}, N, 16, dpct::get_value(beta_c, *handle), result_c_buf_ct{{[0-9]+}}, N, 16, 10);
  // CHECK-NEXT: }
  status = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, alpha_c, x_c, N, 16, y_c, N, 16, beta_c, result_c, N, 16, 10);
  hipblasCgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, alpha_c, x_c, N, 16, y_c, N, 16, beta_c, result_c, N, 16, 10);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, N, 16, y_z_buf_ct{{[0-9]+}}, N, 16, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, N, 16, 10));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, dpct::get_value(alpha_z, *handle), x_z_buf_ct{{[0-9]+}}, N, 16, y_z_buf_ct{{[0-9]+}}, N, 16, dpct::get_value(beta_z, *handle), result_z_buf_ct{{[0-9]+}}, N, 16, 10);
  // CHECK-NEXT: }
  status = hipblasZgemmStridedBatched(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha_z, x_z, N, 16, y_z, N, 16, beta_z, result_z, N, 16, 10);
  hipblasZgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, alpha_z, x_z, N, 16, y_z, N, 16, beta_z, result_z, N, 16, 10);

  const hipComplex** x_c_array = 0;
  const hipComplex** y_c_array = 0;
  hipComplex** result_c_array = 0;
  const hipDoubleComplex** x_z_array = 0;
  const hipDoubleComplex** y_z_array = 0;
  hipDoubleComplex** result_z_array = 0;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasCgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasCgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasZgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasZgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  status = hipblasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  hipblasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  status = hipblasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  hipblasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);

  hipComplex* A_c = 0;
  hipDoubleComplex* A_z = 0;
  hipComplex* B_c = 0;
  hipDoubleComplex* B_z = 0;
  hipComplex* C_c = 0;
  hipDoubleComplex* C_z = 0;


  int ldb = 10;
  int ldc = 10;


  const float alpha_s = 1;
  const float beta_s = 1;
  const double beta_d = 0;

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, m, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, m, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = cublasCgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, m, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, m, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = cublasZgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  int side0 = 0;
  int side1 = 1;
  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(*handle, (oneapi::mkl::side)side1, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  hipblasCsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);
  hipblasZsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT: }
  status = hipblasCtrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_c, A_c, lda, B_c, ldb);
  hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_c, A_c, lda, B_c, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::right, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT: }
  status = hipblasZtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_z, A_z, lda, B_z, ldb);
  hipblasZtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_c, *handle), C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasChemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasChemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, dpct::get_value(beta_z, *handle), C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZhemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZhemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, alpha_s, A_c_buf_ct{{[0-9]+}}, lda, beta_s, C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::herk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, alpha_s, A_c_buf_ct{{[0-9]+}}, lda, beta_s, C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);
  hipblasCherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_d, *handle), A_z_buf_ct{{[0-9]+}}, lda, beta_d, C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::herk(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_d, *handle), A_z_buf_ct{{[0-9]+}}, lda, beta_d, C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);
  hipblasZherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, beta_s, C_c_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2k(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, B_c_buf_ct{{[0-9]+}}, ldb, beta_s, C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasCher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);
  hipblasCher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(*handle, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, beta_d, C_z_buf_ct{{[0-9]+}}, ldc));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2k(*handle, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb, beta_d, C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  status = hipblasZher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
  hipblasZher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);

  // CHECK: {
  // CHECK-NEXT: auto transpose_ct{{[0-9]+}} = foo();
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(*handle, (oneapi::mkl::side)foo(), foo()==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, (int)transpose_ct{{[0-9]+}}==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)transpose_ct{{[0-9]+}}, (oneapi::mkl::diag)foo(), m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, B_z_buf_ct{{[0-9]+}}, ldb);
  // CHECK-NEXT: }
  hipblasZtrsm(handle, (hipblasSideMode_t)foo(), (hipblasFillMode_t)foo(), (hipblasOperation_t)foo(), (hipblasDiagType_t)foo(), m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: {
  // CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(A_c);
  // CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(B_c);
  // CHECK-NEXT: auto C_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(C_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::omatadd(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_c, *handle), A_c_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_c, *handle), B_c_buf_ct{{[0-9]+}}, ldb, C_c_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  hipblasCgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);

  // CHECK: {
  // CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(A_z);
  // CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(B_z);
  // CHECK-NEXT: auto C_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(C_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::omatadd(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_z, *handle), A_z_buf_ct{{[0-9]+}}, lda, dpct::get_value(beta_z, *handle), B_z_buf_ct{{[0-9]+}}, ldb, C_z_buf_ct{{[0-9]+}}, ldc);
  // CHECK-NEXT: }
  hipblasZgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);
}

