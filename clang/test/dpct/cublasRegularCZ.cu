// RUN: dpct --format-range=none --usm-level=none -out-root %T/cublasRegularCZ %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasRegularCZ/cublasRegularCZ.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>
#include <hipblas.h>

int foo();

int main(){
  hipblasStatus_t status;
  hipblasHandle_t handle;

  int* result = 0;
  float* result_f = 0;
  double* result_d = 0;
  hipComplex* x_c = 0;
  hipDoubleComplex* x_z = 0;

  int incx = 1;
  int incy = 1;
  int n = 10;

  //level 1
  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIcamax(handle, n, x_c, incx, result);
  hipblasIcamax(handle, n, x_c, incx, result);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIzamax(handle, n, x_z, incx, result);
  hipblasIzamax(handle, n, x_z, incx, result);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIcamin(handle, n, x_c, incx, result);
  hipblasIcamin(handle, n, x_c, incx, result);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIzamin(handle, n, x_z, incx, result);
  hipblasIzamin(handle, n, x_z, incx, result);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasScasum(handle, n, x_c, incx, result_f);
  hipblasScasum(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasDzasum(handle, n, x_z, incx, result_d);
  hipblasDzasum(handle, n, x_z, incx, result_d);

  hipComplex* alpha_c = 0;
  hipComplex* beta_c = 0;
  hipDoubleComplex* alpha_z = 0;
  hipDoubleComplex* beta_z = 0;
  float* alpha_f = 0;
  double* alpha_d = 0;
  hipComplex* y_c = 0;
  hipDoubleComplex* y_z = 0;

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy);
  status = hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);
  hipblasCaxpy(handle, n, alpha_c, x_c, incx, y_c, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy);
  status = hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);
  hipblasZaxpy(handle, n, alpha_z, x_z, incx, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy);
  status = hipblasCcopy(handle, n, x_c, incx, y_c, incy);
  hipblasCcopy(handle, n, x_c, incx, y_c, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy);
  status = hipblasZcopy(handle, n, x_z, incx, y_z, incy);
  hipblasZcopy(handle, n, x_z, incx, y_z, incy);

  hipComplex* result_c = 0;
  hipDoubleComplex* result_z = 0;

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotu(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);
  hipblasCdotc(handle, n, x_c, incx, y_c, incy, result_c);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotu(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);
  hipblasZdotc(handle, n, x_z, incx, y_z, incy, result_z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_f);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasScnrm2(handle, n, x_c, incx, result_f);
  hipblasScnrm2(handle, n, x_c, incx, result_f);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasDznrm2(handle, n, x_z, incx, result_d);
  hipblasDznrm2(handle, n, x_z, incx, result_d);

  float* c_f = 0;
  float* s_f = 0;
  double* c_d = 0;
  double* s_d = 0;
  hipComplex* c_c = 0;
  hipComplex* s_c = 0;
  hipDoubleComplex* c_z = 0;
  hipDoubleComplex* s_z = 0;

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::get_value(c_f, handle->get_queue()), dpct::get_value(s_f, handle->get_queue())));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::get_value(c_f, handle->get_queue()), dpct::get_value(s_f, handle->get_queue()));
  status = hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);
  hipblasCsrot(handle, n, x_c, incx, y_c, incy, c_f, s_f);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::get_value(c_d, handle->get_queue()), dpct::get_value(s_d, handle->get_queue())));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::get_value(c_d, handle->get_queue()), dpct::get_value(s_d, handle->get_queue()));
  status = hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);
  hipblasZdrot(handle, n, x_z, incx, y_z, incy, c_d, s_d);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::get_value(c_f, handle->get_queue()), dpct::get_value(s_c, handle->get_queue())));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy, dpct::get_value(c_f, handle->get_queue()), dpct::get_value(s_c, handle->get_queue()));
  status = hipblasCrot(handle, n, x_c, incx, y_c, incy, c_f, s_c);
  hipblasCrot(handle, n, x_c, incx, y_c, incy, c_f, s_c);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::get_value(c_d, handle->get_queue()), dpct::get_value(s_z, handle->get_queue())));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy, dpct::get_value(c_d, handle->get_queue()), dpct::get_value(s_z, handle->get_queue()));
  status = hipblasZrot(handle, n, x_z, incx, y_z, incy, c_d, s_z);
  hipblasZrot(handle, n, x_z, incx, y_z, incy, c_d, s_z);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_inout res_wrapper_ct1(handle->get_queue(), x_c);
  // CHECK-NEXT: dpct::blas::wrapper_float2_inout res_wrapper_ct2(handle->get_queue(), y_c);
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct3(handle->get_queue(), c_f);
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct4(handle->get_queue(), s_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct1.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct2.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct3.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_inout res_wrapper_ct1(handle->get_queue(), x_c);
  // CHECK-NEXT: dpct::blas::wrapper_float2_inout res_wrapper_ct2(handle->get_queue(), y_c);
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct3(handle->get_queue(), c_f);
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct4(handle->get_queue(), s_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct1.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct2.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct3.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasCrotg(handle, x_c, y_c, c_f, s_c);
  hipblasCrotg(handle, x_c, y_c, c_f, s_c);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_inout res_wrapper_ct1(handle->get_queue(), x_z);
  // CHECK-NEXT: dpct::blas::wrapper_double2_inout res_wrapper_ct2(handle->get_queue(), y_z);
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct3(handle->get_queue(), c_d);
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct4(handle->get_queue(), s_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct1.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct2.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct3.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_inout res_wrapper_ct1(handle->get_queue(), x_z);
  // CHECK-NEXT: dpct::blas::wrapper_double2_inout res_wrapper_ct2(handle->get_queue(), y_z);
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct3(handle->get_queue(), c_d);
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct4(handle->get_queue(), s_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct1.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct2.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct3.get_ptr())), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasZrotg(handle, x_z, y_z, c_d, s_z);
  hipblasZrotg(handle, x_z, y_z, c_d, s_z);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx);
  status = hipblasCscal(handle, n, alpha_c, x_c, incx);
  hipblasCscal(handle, n, alpha_c, x_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx);
  status = hipblasZscal(handle, n, alpha_z, x_z, incx);
  hipblasZscal(handle, n, alpha_z, x_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_f, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_f, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx);
  status = hipblasCsscal(handle, n, alpha_f, x_c, incx);
  hipblasCsscal(handle, n, alpha_f, x_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx);
  status = hipblasZdscal(handle, n, alpha_d, x_z, incx);
  hipblasZdscal(handle, n, alpha_d, x_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy);
  status = hipblasCswap(handle, n, x_c, incx, y_c, incy);
  hipblasCswap(handle, n, x_c, incx, y_c, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy);
  status = hipblasZswap(handle, n, x_z, incx, y_z, incy);
  hipblasZswap(handle, n, x_z, incx, y_z, incy);

  //level 2
  int m=0;
  int kl=0;
  int ku=0;
  int lda = 10;
  int trans0 = 0;
  int trans1 = 1;
  int trans2 = 2;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), dpct::get_transpose(trans0), m, n, kl, ku, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), oneapi::mkl::transpose::nontrans, m, n, kl, ku, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy);
  status = hipblasCgbmv(handle, (hipblasOperation_t)trans0, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), dpct::get_transpose(trans1), m, n, kl, ku, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), oneapi::mkl::transpose::nontrans, m, n, kl, ku, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy);
  status = hipblasZgbmv(handle, (hipblasOperation_t)trans1, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgbmv(handle, HIPBLAS_OP_N, m, n, kl, ku, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), dpct::get_transpose(trans2), m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(handle->get_queue(), oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy);
  status = hipblasCgemv(handle, (hipblasOperation_t)trans2, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasCgemv(handle, HIPBLAS_OP_N, m, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemv(handle->get_queue(), oneapi::mkl::transpose::nontrans, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy);
  status = hipblasZgemv(handle, (hipblasOperation_t)0, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, m, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(handle->get_queue(), m, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::geru(handle->get_queue(), m, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgeru(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gerc(handle->get_queue(), m, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gerc(handle->get_queue(), m, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCgerc(handle, m, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(handle->get_queue(), m, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::geru(handle->get_queue(), m, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgeru(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gerc(handle->get_queue(), m, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gerc(handle->get_queue(), m, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZgerc(handle, m, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  int k = 1;
  int fill0 = 0;
  int fill1 = 1;
  int diag0 = 0;
  int diag1 = 1;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, (oneapi::mkl::diag)diag0, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx);
  status = hipblasCtbmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)1, (hipblasDiagType_t)diag0, n, k, x_c, lda, result_c, incx);
  hipblasCtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_c, lda, result_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), fill1 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, (oneapi::mkl::diag)diag1, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx);
  status = hipblasZtbmv(handle, (hipblasFillMode_t)fill1, (hipblasOperation_t)2, (hipblasDiagType_t)diag1, n, k, x_z, lda, result_z, incx);
  hipblasZtbmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, k, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(handle->get_queue(), oneapi::mkl::uplo::lower, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::nonunit,  n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit,  n, k, x_c_buf_ct{{[0-9]+}}, lda, result_c_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasCtbsv(handle, (hipblasFillMode_t)0, (hipblasOperation_t)trans0, (hipblasDiagType_t)0,  n, k, x_c, lda, result_c, incx);
  hipblasCtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_c, lda, result_c, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbsv(handle->get_queue(), oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, oneapi::mkl::diag::unit,  n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tbsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit,  n, k, x_z_buf_ct{{[0-9]+}}, lda, result_z_buf_ct{{[0-9]+}}, incx);
  // CHECK-NEXT: }
  status = hipblasZtbsv(handle, (hipblasFillMode_t)1, (hipblasOperation_t)trans0, (hipblasDiagType_t)1,  n, k, x_z, lda, result_z, incx);
  hipblasZtbsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT,  n, k, x_z, lda, result_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx);
  status = hipblasCtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx);
  status = hipblasZtpmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx);
  status = hipblasCtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, result_c, incx);
  hipblasCtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, result_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx);
  status = hipblasZtpsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, result_z, incx);
  hipblasZtpsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, result_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx);
  status = hipblasCtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx);
  status = hipblasZtrmv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrmv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), incx);
  status = hipblasCtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_c, lda, result_c, incx);
  hipblasCtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_c, lda, result_c, incx);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsv(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), incx);
  status = hipblasZtrsv(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, n, x_z, lda, result_z, incx);
  hipblasZtrsv(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, n, x_z, lda, result_z, incx);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemv(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChemv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemv(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhemv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhemv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, k, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), oneapi::mkl::uplo::lower, n, k, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);
  hipblasChbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_c, x_c, lda, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, k, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), oneapi::mkl::uplo::lower, n, k, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhbmv(handle, (hipblasFillMode_t)fill0, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);
  hipblasZhbmv(handle, HIPBLAS_FILL_MODE_LOWER, n, k, alpha_z, x_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpmv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpmv(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, x_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), y_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasChpmv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);
  hipblasChpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, x_c, incx, beta_c, y_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpmv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpmv(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, x_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), y_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZhpmv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);
  hipblasZhpmv(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, x_z, incx, beta_z, y_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_f, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_f, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c, lda);
  hipblasCher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_d, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_d, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z, lda);
  hipblasZher(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCher2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZher2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZher2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_f, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_f, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr(handle, (hipblasFillMode_t)fill0, n, alpha_f, x_c, incx, result_c);
  hipblasChpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_f, x_c, incx, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_d, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_d, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr(handle, (hipblasFillMode_t)fill0, n, alpha_d, x_z, incx, result_z);
  hipblasZhpr(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_d, x_z, incx, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr2(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasChpr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c);
  hipblasChpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_c, x_c, incx, y_c, incy, result_c);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hpr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hpr2(handle->get_queue(), oneapi::mkl::uplo::lower, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}});
  // CHECK-NEXT: }
  status = hipblasZhpr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z);
  hipblasZhpr2(handle, HIPBLAS_FILL_MODE_LOWER, n, alpha_z, x_z, incx, y_z, incy, result_z);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, y_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), result_c_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, lda, y_c_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_c, handle->get_queue()), result_c_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasCsymv(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, lda, y_c, incx, beta_c, result_c, incy);
  hipblasCsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, lda, y_c, incx, beta_c, result_c, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, y_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), result_z_buf_ct{{[0-9]+}}, incy));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, lda, y_z_buf_ct{{[0-9]+}}, incx, dpct::get_value(beta_z, handle->get_queue()), result_z_buf_ct{{[0-9]+}}, incy);
  // CHECK-NEXT: }
  status = hipblasZsymv(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, lda, y_z, incx, beta_z, result_z, incy);
  hipblasZsymv(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, lda, y_z, incx, beta_z, result_z, incy);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCsyr(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, result_c, lda);
  hipblasCsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, incx, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZsyr(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, result_z, lda);
  hipblasZsyr(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, incx, result_z, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_c);
  // CHECK-NEXT: auto y_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_c);
  // CHECK-NEXT: auto result_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_c, handle->get_queue()), x_c_buf_ct{{[0-9]+}}, incx, y_c_buf_ct{{[0-9]+}}, incy, result_c_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasCsyr2(handle, (hipblasFillMode_t)fill0, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);
  hipblasCsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_c, x_c, incx, y_c, incy, result_c, lda);

  // CHECK: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda));
  // CHECK-NEXT: }
  // CHECK-NEXT: {
  // CHECK-NEXT: auto x_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(x_z);
  // CHECK-NEXT: auto y_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(y_z);
  // CHECK-NEXT: auto result_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(handle->get_queue(), oneapi::mkl::uplo::upper, n, dpct::get_value(alpha_z, handle->get_queue()), x_z_buf_ct{{[0-9]+}}, incx, y_z_buf_ct{{[0-9]+}}, incy, result_z_buf_ct{{[0-9]+}}, lda);
  // CHECK-NEXT: }
  status = hipblasZsyr2(handle, (hipblasFillMode_t)fill0, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);
  hipblasZsyr2(handle, HIPBLAS_FILL_MODE_UPPER, n, alpha_z, x_z, incx, y_z, incy, result_z, lda);


  // level 3
  int N = 100;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans0), N, N, N, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), N, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), N, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);
  hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_c, x_c, N, y_c, N, beta_c, result_c, N);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans0), N, N, N, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), N, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), N, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), N, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), N, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, alpha_z, x_z, N, y_z, N, beta_z, result_z, N);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::trans, N, N, N, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), N, 16, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), N, 16, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), N, 16, 10, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), N, 16, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), N, 16, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(result_c)), N, 16, 10, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_T, N, N, N, alpha_c, x_c, N, 16, y_c, N, 16, beta_c, result_c, N, 16, 10);
  hipblasCgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, alpha_c, x_c, N, 16, y_c, N, 16, beta_c, result_c, N, 16, 10);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), N, 16, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), N, 16, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), N, 16, 10, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), N, 16, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), N, 16, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(result_z)), N, 16, 10, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZgemmStridedBatched(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha_z, x_z, N, 16, y_z, N, 16, beta_z, result_z, N, 16, 10);
  hipblasZgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, alpha_z, x_z, N, 16, y_z, N, 16, beta_z, result_z, N, 16, 10);

  const hipComplex** x_c_array = 0;
  const hipComplex** y_c_array = 0;
  hipComplex** result_c_array = 0;
  const hipDoubleComplex** x_z_array = 0;
  const hipDoubleComplex** y_z_array = 0;
  hipDoubleComplex** result_z_array = 0;
  hipblasOperation_t trans3 = HIPBLAS_OP_N;

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasCgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasCgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasZgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = cublasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cublasZgemmBatched is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cublasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  status = hipblasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  hipblasCgemmBatched(handle, trans3, trans3, N, N, N, alpha_c, x_c_array, N, y_c_array, N, beta_c, result_c_array, N, 10);
  status = hipblasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);
  hipblasZgemmBatched(handle, trans3, trans3, N, N, N, alpha_z, x_z_array, N, y_z_array, N, beta_z, result_z_array, N, 10);

  hipComplex* A_c = 0;
  hipDoubleComplex* A_z = 0;
  hipComplex* B_c = 0;
  hipDoubleComplex* B_z = 0;
  hipComplex* C_c = 0;
  hipDoubleComplex* C_z = 0;


  int ldb = 10;
  int ldc = 10;


  const float alpha_s = 1;
  const float beta_s = 1;
  const double beta_d = 0;

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans0), m, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, oneapi::mkl::compute_mode::complex_3m));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, m, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, oneapi::mkl::compute_mode::complex_3m);
  status = cublasCgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  cublasCgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans0), m, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, oneapi::mkl::compute_mode::complex_3m));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, m, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, oneapi::mkl::compute_mode::complex_3m);
  status = cublasZgemm3m(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans0, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  cublasZgemm3m(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  int side0 = 0;
  int side1 = 1;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCsymm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), (oneapi::mkl::side)side1, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZsymm(handle, (hipblasSideMode_t)side1, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsymm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  hipblasCsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syrk(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZsyrk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);
  hipblasZsyrk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasCsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZsyr2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZsyr2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), oneapi::mkl::side::left, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCtrsm(handle, (hipblasSideMode_t)0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_c, A_c, lda, B_c, ldb);
  hipblasCtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_c, A_c, lda, B_c, ldb);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), oneapi::mkl::side::right, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::nonunit, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZtrsm(handle, (hipblasSideMode_t)1, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, m, n, alpha_z, A_z, lda, B_z, ldb);
  hipblasZtrsm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasChemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  hipblasChemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::hemm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZhemm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  hipblasZhemm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, alpha_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, beta_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::herk(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, alpha_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, beta_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);
  hipblasCherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, &alpha_s, A_c, lda, &beta_s, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, beta_d, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::herk(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, beta_d, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZherk(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);
  hipblasZherk(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_d, A_z, lda, &beta_d, C_z, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, beta_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2k(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, beta_s, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasCher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);
  hipblasCher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_c, A_c, lda, B_c, ldb, &beta_s, C_c, ldc);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, beta_d, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true)));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::her2k(handle->get_queue(), oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, beta_d, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  status = hipblasZher2k(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);
  hipblasZher2k(handle, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, n, k, alpha_z, A_z, lda, B_z, ldb, &beta_d, C_z, ldc);

  // CHECK: oneapi::mkl::blas::column_major::trsm(handle->get_queue(), (oneapi::mkl::side)foo(), foo() == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(foo()), (oneapi::mkl::diag)foo(), m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::blas::deduce_compute_mode(std::nullopt, handle->get_math_mode(), true));
  hipblasZtrsm(handle, (hipblasSideMode_t)foo(), (hipblasFillMode_t)foo(), (hipblasOperation_t)foo(), (hipblasDiagType_t)foo(), m, n, alpha_z, A_z, lda, B_z, ldb);

  // CHECK: oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc);
  hipblasCgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);

  // CHECK: oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc);
  hipblasZgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);
}

