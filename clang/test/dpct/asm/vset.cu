
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/asm/vset %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm/vset/vset.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl  %T/asm/vset/vset.dp.cpp -o %T/asm/vset/vset.dp.o %}

// clang-format off
__global__ void vset(unsigned *d) {
  unsigned a, b;
  int e, f;

  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::equal_to<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::not_equal_to<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::less<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::less_equal<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::greater<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, std::greater_equal<>());
  asm("vset.s32.s32.eq %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  asm("vset.s32.s32.ne %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  asm("vset.s32.s32.lt %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  asm("vset.s32.s32.le %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  asm("vset.s32.s32.gt %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  asm("vset.s32.s32.ge %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(0));
  
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::equal_to<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::not_equal_to<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::less<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::less_equal<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::greater<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, std::greater_equal<>());
  asm("vset.u32.s32.eq %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));
  asm("vset.u32.s32.ne %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));
  asm("vset.u32.s32.lt %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));
  asm("vset.u32.s32.le %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));
  asm("vset.u32.s32.gt %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));
  asm("vset.u32.s32.ge %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(0));

  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::equal_to<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::not_equal_to<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::less<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::less_equal<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::greater<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, std::greater_equal<>());
  asm("vset.s32.u32.eq %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));
  asm("vset.s32.u32.ne %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));
  asm("vset.s32.u32.lt %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));
  asm("vset.s32.u32.le %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));
  asm("vset.s32.u32.gt %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));
  asm("vset.s32.u32.ge %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(0));

  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::equal_to<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::not_equal_to<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::less<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::less_equal<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::greater<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, uint32_t>(a, b, std::greater_equal<>());
  asm("vset.u32.u32.eq %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));
  asm("vset.u32.u32.ne %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));
  asm("vset.u32.u32.lt %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));
  asm("vset.u32.u32.le %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));
  asm("vset.u32.u32.gt %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));
  asm("vset.u32.u32.ge %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(b), "r"(0));

  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::equal_to<>(), sycl::plus<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::not_equal_to<>(), sycl::plus<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::less<>(), sycl::plus<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::less_equal<>(), sycl::plus<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::greater<>(), sycl::plus<>());
  // CHECK: *d = dpct::extend_compare<int32_t, int32_t>(e, f, a, std::greater_equal<>(), sycl::plus<>());
  asm("vset.s32.s32.eq.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));
  asm("vset.s32.s32.ne.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));
  asm("vset.s32.s32.lt.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));
  asm("vset.s32.s32.le.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));
  asm("vset.s32.s32.gt.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));
  asm("vset.s32.s32.ge.add %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(f), "r"(a));

  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::equal_to<>(), sycl::minimum<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::not_equal_to<>(), sycl::minimum<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::less<>(), sycl::minimum<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::less_equal<>(), sycl::minimum<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::greater<>(), sycl::minimum<>());
  // CHECK: *d = dpct::extend_compare<int32_t, uint32_t>(e, a, b, std::greater_equal<>(), sycl::minimum<>());
  asm("vset.s32.u32.eq.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));
  asm("vset.s32.u32.ne.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));
  asm("vset.s32.u32.lt.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));
  asm("vset.s32.u32.le.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));
  asm("vset.s32.u32.gt.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));
  asm("vset.s32.u32.ge.min %0, %1, %2, %3;" : "=r"(*d) : "r"(e), "r"(a), "r"(b));

  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::equal_to<>(), sycl::maximum<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::not_equal_to<>(), sycl::maximum<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::less<>(), sycl::maximum<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::less_equal<>(), sycl::maximum<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::greater<>(), sycl::maximum<>());
  // CHECK: *d = dpct::extend_compare<uint32_t, int32_t>(a, f, a, std::greater_equal<>(), sycl::maximum<>());
  asm("vset.u32.s32.eq.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
  asm("vset.u32.s32.ne.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
  asm("vset.u32.s32.lt.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
  asm("vset.u32.s32.le.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
  asm("vset.u32.s32.gt.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
  asm("vset.u32.s32.ge.max %0, %1, %2, %3;" : "=r"(*d) : "r"(a), "r"(f), "r"(a));
}
