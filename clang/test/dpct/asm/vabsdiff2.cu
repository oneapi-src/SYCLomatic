
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/asm/vabsdiff2 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm/vabsdiff2/vabsdiff2.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/vabsdiff2/vabsdiff2.dp.cpp -o %T/vabsdiff2/vabsdiff2.dp.o %}

__global__ void vabsdiff2() {
  int a, b, c, d;

  // CHECK: d = dpct::extend_vabsdiff2<int32_t, int32_t, uint32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vabsdiff2_sat<int32_t, int32_t, uint32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vabsdiff2_add<int32_t, int32_t, uint32_t>(a, b, c);
  // clang-format off
  asm("vabsdiff2.s32.s32.u32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vabsdiff2.s32.s32.u32.sat %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vabsdiff2.s32.s32.u32.add %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  // clang-format on
}

int main() {
  vabsdiff2<<<1, 1>>>();
  return 0;
}
