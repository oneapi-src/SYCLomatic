// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/ld %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/ld/ld.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/ld/ld.dp.cpp -o %T/ld/ld.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>

/*
.ss =                       { .const, .global, .local, .param, .shared };
.type =                     { .b8, .b16, .b32, .b64, .b128, 
                              .u8, .u16, .u32, .u64,
                              .s8, .s16, .s32, .s64,
                              .f32, .f64 };
Current only support the form likes "ld.ss.type" now.
*/

__global__ void ld(int *arr) {
  int a, b, c;
  unsigned long long d;
  // CHECK: a = *arr;
  asm volatile ("ld.global.s32 %0, [%1];" : "=r"(a) : "l"(arr));
  // CHECK: b = *((uint32_t *)(uintptr_t)arr);
  asm volatile ("ld.global.u32 %0, [%1];" : "=r"(b) : "l"(arr));
  // CHECK: c = *((uint32_t *)((uintptr_t)arr + 4));
  asm volatile ("ld.global.u32 %0, [%1 + 4];" : "=r"(c) : "l"(arr));
  // CHECK: d = *((uint64_t *)((uintptr_t)arr + 8));
  asm volatile ("ld.global.u64 %0, [%1 + 8];" : "=l"(d) : "l"(arr));
}

__device__ void shared_address_load32(uint32_t addr, uint32_t &val) {
  // CHECK: {
  // CHECK:   val = *((uint32_t *)(uintptr_t)addr);
  // CHECK: } 
  asm volatile("{ld.shared.b32 %0, [%1];}" : : "r"(val), "r"(addr) : "memory"); 
}

// clang-format on
