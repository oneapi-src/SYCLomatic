// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/popc %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/popc/popc.dp.cpp

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void popc() {
  int x = 1;
  int32_t i32;
  int64_t i64;
  // CHECK: i32 = sycl::popcount<uint32_t>(x);
  asm("popc.b32 %0, %1;" : "=r"(i32) : "r"(x));

  // CHECK: i64 = sycl::popcount<uint64_t>(x);
  asm("popc.b64 %0, %1;" : "=r"(i64) : "r"(x));
}

// clang-format on
