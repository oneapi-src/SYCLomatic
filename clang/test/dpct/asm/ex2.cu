// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/ex2 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/ex2/ex2.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/ex2/ex2.dp.cpp -o %T/ex2/ex2.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void ex2() {
  float f32;
  // CHECK: f32 = sycl::pow<float>(2, 3.4f);
  asm("ex2.approx.f32 %0, %1;" : "=f"(f32) : "f"(3.4f));
  
  // CHECK: f32 = sycl::pow<float>(2, 3.4f);
  asm("ex2.approx.ftz.f32 %0, %1;" : "=f"(f32) : "f"(3.4f));
}

// clang-format on
