// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/vavrg4 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/vavrg4/vavrg4.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/vavrg4/vavrg4.dp.cpp -o %T/vavrg4/vavrg4.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>

__global__ void varvg4() {
  int a, b, c, d;
  // CHECK: d = dpct::extend_vavrg4<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vavrg4_sat<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vavrg4_add<int32_t, uint32_t, int32_t>(a, b, c);
  asm("vavrg4.s32.u32.s32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vavrg4.s32.u32.s32.sat %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vavrg4.s32.u32.s32.add %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
}

// clang-format on
