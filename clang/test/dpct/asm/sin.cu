// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/sin %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/sin/sin.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/sin/sin.dp.cpp -o %T/sin/sin.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void sin() {
  float f32;

  // CHECK: f32 = sycl::sin(1.0f);
  asm("sin.approx.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
  
  // CHECK: f32 = sycl::sin(1.0f);
  asm("sin.approx.ftz.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
}

// clang-format on
