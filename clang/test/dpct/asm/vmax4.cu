
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/asm/vmax4 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm/vmax4/vmax4.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/vmax4/vmax4.dp.cpp -o %T/vmax4/vmax4.dp.o %}

__global__ void vmax4() {
  int a, b, c, d;

  // CHECK: d = dpct::extend_vmax4<uint32_t, uint32_t, uint32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vmax4_sat<uint32_t, uint32_t, uint32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vmax4_add<uint32_t, uint32_t, uint32_t>(a, b, c);
  // clang-format off
  asm("vmax4.u32.u32.u32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vmax4.u32.u32.u32.sat %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vmax4.u32.u32.u32.add %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  // clang-format on
}

int main() {
  vmax4<<<1, 1>>>();
  return 0;
}
