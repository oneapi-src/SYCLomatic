// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/lg2 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/lg2/lg2.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/lg2/lg2.dp.cpp -o %T/lg2/lg2.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void lg2() {
  float f32;
  // CHECK: f32 = sycl::log2(1.0f);
  asm("lg2.approx.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
  
  // CHECK: f32 = sycl::log2(1.0f);
  asm("lg2.approx.ftz.f32 %0, %1;" : "=f"(f32) : "f"(1.0f));
}

// clang-format on
