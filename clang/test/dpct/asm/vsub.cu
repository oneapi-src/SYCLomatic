// RUN: dpct -out-root %T/vsub %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/vsub/vsub.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/vsub/vsub.dp.cpp -o %T/vsub/vsub.dp.o %}


// clang-format off

#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void vsub() {
  int a, b, c, d;

  // CHECK: a = dpct::extend_sub<int32_t>(b, c);
  asm("vsub.s32.u32.s32 %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_sub<uint32_t>(b, c);
  asm("vsub.u32.u32.s32 %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_sub_sat<int32_t>(b, c);
  asm("vsub.s32.u32.s32.sat %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_sub_sat<uint32_t>(b, c);
  asm("vsub.u32.u32.s32.sat %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_sub_sat<int32_t>(b, c, d, sycl::plus<>());
  asm("vsub.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));
  
  // CHECK: a = dpct::extend_sub_sat<int32_t>(b, c, d, sycl::minimum<>());
  asm("vsub.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));

  // CHECK: a = dpct::extend_sub_sat<int32_t>(b, c, d, sycl::maximum<>());
  asm("vsub.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));
}

// clang-format on
