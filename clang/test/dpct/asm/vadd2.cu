
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/asm/vadd2 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm/vadd2/vadd2.dp.cpp

__global__ void vadd2() {
  int a, b, c, d;

  // CHECK: d = dpct::extend_vadd2<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vadd2_sat<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vadd2_add<int32_t, uint32_t, int32_t>(a, b, c);
  // clang-format off
  asm("vadd2.s32.u32.s32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vadd2.s32.u32.s32.sat %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vadd2.s32.u32.s32.add %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  // clang-format on
}

int main() {
  vadd2<<<1, 1>>>();
  return 0;
}
