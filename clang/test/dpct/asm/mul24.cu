// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/mul24 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/mul24/mul24.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/mul24/mul24.dp.cpp -o %T/mul24/mul24.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void mul24() {
  int x = 1, y = 2;
  int32_t i32;
  uint32_t u32;

  // CHECK: i32 = sycl::mul24((int32_t)x, (int32_t)y);
  asm("mul24.lo.s32 %0, %1, %2;" : "=r"(i32) : "r"(x), "r"(y));

  // CHECK: u32 = sycl::mul24((uint32_t)x, (uint32_t)y);
  asm("mul24.lo.u32 %0, %1, %2;" : "=r"(u32) : "r"(x), "r"(y));

  // CHECK: DPCT1053:{{.*}}: Migration of device assembly code is not supported.
  asm("mul24.hi.s32 %0, %1, %2;" : "=r"(i32) : "r"(x), "r"(y));
  
  // CHECK: DPCT1053:{{.*}}: Migration of device assembly code is not supported.
  asm("mul24.hi.u32 %0, %1, %2;" : "=r"(u32) : "r"(x), "r"(y));
}

// clang-format on
