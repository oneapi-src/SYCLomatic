// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/xor %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/xor/xor.dp.cpp

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void bitwise_xor() {
  int x = 1, y = 2;
  uint8_t u8;
  uint16_t u16;
  uint32_t u32;
  uint64_t u64;

  // CHECK: u8 = x ^ y;
  asm("xor.pred %0, %1, %2;" : "=r"(u8) : "r"(x), "r"(y));

  // CHECK: u16 = x ^ y;
  asm("xor.b16 %0, %1, %2;" : "=r"(u16) : "r"(x), "r"(y));

  // CHECK: u32 = x ^ y;
  asm("xor.b32 %0, %1, %2;" : "=r"(u32) : "r"(x), "r"(y));

  // CHECK: u64 = x ^ y;
  asm("xor.b64 %0, %1, %2;" : "=r"(u64) : "r"(x), "r"(y));
}

// clang-format on
