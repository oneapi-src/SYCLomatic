// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/div %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/div/div.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/div/div.dp.cpp -o %T/div/div.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void div() {
  int x = 1, y = 2;
  int16_t i16;
  uint16_t u16;
  int32_t i32;
  uint32_t u32;
  int64_t i64;
  uint64_t u64;

  // CHECK: i16 = x / y;
  asm("div.s16 %0, %1, %2;" : "=r"(i16) : "r"(x), "r"(y));

  // CHECK: u16 = x / y;
  asm("div.u16 %0, %1, %2;" : "=r"(u16) : "r"(x), "r"(y));

  // CHECK: i32 = x / y;
  asm("div.s32 %0, %1, %2;" : "=r"(i32) : "r"(x), "r"(y));

  // CHECK: u32 = x / y;
  asm("div.u32 %0, %1, %2;" : "=r"(u32) : "r"(x), "r"(y));

  // CHECK: i64 = x / y;
  asm("div.s64 %0, %1, %2;" : "=r"(i64) : "r"(x), "r"(y));

  // CHECK: u64 = x / y;
  asm("div.u64 %0, %1, %2;" : "=r"(u64) : "r"(x), "r"(y));
}

// clang-format on
