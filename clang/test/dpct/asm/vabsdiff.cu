// RUN: dpct -out-root %T/vabsdiff %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/vabsdiff/vabsdiff.dp.cpp


// clang-format off

#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void vabsdiff() {
  int a, b, c, d;

  // CHECK: a = dpct::extend_absdiff<int32_t>(b, c);
  asm("vabsdiff.s32.u32.s32 %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_absdiff<uint32_t>(b, c);
  asm("vabsdiff.u32.u32.s32 %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_absdiff_sat<int32_t>(b, c);
  asm("vabsdiff.s32.u32.s32.sat %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_absdiff_sat<uint32_t>(b, c);
  asm("vabsdiff.u32.u32.s32.sat %0, %1, %2;" : "=r"(a) : "r"(b), "r"(c));

  // CHECK: a = dpct::extend_absdiff_sat<int32_t>(b, c, d, sycl::plus<>());
  asm("vabsdiff.s32.u32.s32.sat.add %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));
  
  // CHECK: a = dpct::extend_absdiff_sat<int32_t>(b, c, d, sycl::minimum<>());
  asm("vabsdiff.s32.u32.s32.sat.min %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));

  // CHECK: a = dpct::extend_absdiff_sat<int32_t>(b, c, d, sycl::maximum<>());
  asm("vabsdiff.s32.u32.s32.sat.max %0, %1, %2, %3;" : "=r"(a) : "r"(b), "r"(c), "r"(d));
}

// clang-format on
