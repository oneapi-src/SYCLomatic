// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/fence %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/fence/fence.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/fence/fence.dp.cpp -o %T/fence/fence.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void fence() {

  // CHECK: sycl::atomic_fence(sycl::memory_order::acq_rel, sycl::memory_scope::device);
  asm("fence.acq_rel.gpu;");
  
}

// clang-format on
