// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/max %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/max/max.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/max/max.dp.cpp -o %T/max/max.dp.o %}

// clang-format off
#include <hip/hip_runtime.h>
#include <cstdint>

__global__ void max() {
  int x = 1, y = 2;
  int16_t i16;
  uint16_t u16;
  int32_t i32;
  uint32_t u32;
  int64_t i64;
  uint64_t u64;
  short2 s16x2, sa{1, 2}, sb{1, 2};
  ushort2 u16x2, ua{1, 2}, ub{1, 2};

  // CHECK: i16 = sycl::max((int16_t)x, (int16_t)y);
  asm("max.s16 %0, %1, %2;" : "=r"(i16) : "r"(x), "r"(y));

  // CHECK: u16 = sycl::max((uint16_t)x, (uint16_t)y);
  asm("max.u16 %0, %1, %2;" : "=r"(u16) : "r"(x), "r"(y));

  // CHECK: i32 = sycl::max((int32_t)x, (int32_t)y);
  asm("max.s32 %0, %1, %2;" : "=r"(i32) : "r"(x), "r"(y));

  // CHECK: u32 = sycl::max((uint32_t)x, (uint32_t)y);
  asm("max.u32 %0, %1, %2;" : "=r"(u32) : "r"(x), "r"(y));

  // CHECK: i64 = sycl::max((int64_t)x, (int64_t)y);
  asm("max.s64 %0, %1, %2;" : "=r"(i64) : "r"(x), "r"(y));

  // CHECK: u64 = sycl::max((uint64_t)x, (uint64_t)y);
  asm("max.u64 %0, %1, %2;" : "=r"(u64) : "r"(x), "r"(y));

  // CHECK: s16x2 = sycl::max((sycl::short2)sa, (sycl::short2)sb);
  asm("max.s16x2 %0, %1, %2;" : "=r"(s16x2) : "r"(sa), "r"(sb));

  // CHECK: u16x2 = sycl::max((sycl::ushort2)ua, (sycl::ushort2)ub);
  asm("max.u16x2 %0, %1, %2;" : "=r"(u16x2) : "r"(ua), "r"(ub));

  // CHECK: i32 = dpct::relu(sycl::max((int32_t)x, (int32_t)y));
  asm("max.s32.relu %0, %1, %2;" : "=r"(i32) : "r"(x), "r"(y));

  // CHECK: s16x2 = dpct::relu(sycl::max((sycl::short2)sa, (sycl::short2)sb));
  asm("max.s16x2.relu %0, %1, %2;" : "=r"(s16x2) : "r"(sa), "r"(sb));
}

// clang-format on
