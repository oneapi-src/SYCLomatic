
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/asm/vadd4 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/asm/vadd4/vadd4.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/vadd4/vadd4.dp.cpp -o %T/vadd4/vadd4.dp.o %}

__global__ void vadd4() {
  int a, b, c, d;

  // CHECK: d = dpct::extend_vadd4<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vadd4_sat<int32_t, uint32_t, int32_t>(a, b, c);
  // CHECK-NEXT: d = dpct::extend_vadd4_add<int32_t, uint32_t, int32_t>(a, b, c);
  // clang-format off
  asm("vadd4.s32.u32.s32 %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vadd4.s32.u32.s32.sat %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  asm("vadd4.s32.u32.s32.add %0, %1, %2, %3;" : "=r"(d) : "r"(a), "r"(b), "r"(c));
  // clang-format on
}

int main() {
  vadd4<<<1, 1>>>();
  return 0;
}
