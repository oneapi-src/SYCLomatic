
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -usm-level=none -out-root %T/memory_data_types %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memory_data_types/memory_data_types.dp.cpp %s
// RUN: %if build_lit %{icpx -c -fsycl %T/memory_data_types/memory_data_types.dp.cpp -o %T/memory_data_types/memory_data_types.dp.o %}

void foo(int *data, int x, int y) {
  // CHECK: dpct::pitched_data p1 = dpct::pitched_data(data, x, x, y);
  hipPitchedPtr p1 = make_hipPitchedPtr(data, x, x, y);

  size_t p1_pitch, p1_x, p1_y;

  // CHECK: data = (int *)p1.get_data_ptr();
  // CHECK-NEXT: p1.set_data_ptr(data);
  // CHECK-NEXT: p1_pitch = p1.get_pitch();
  // CHECK-NEXT: p1.set_pitch(p1_pitch);
  // CHECK-NEXT: p1_x = p1.get_x();
  // CHECK-NEXT: p1.set_x(p1_x);
  // CHECK-NEXT: p1_y = p1.get_y();
  // CHECK-NEXT: p1.set_y(p1_y);
  data = (int *)p1.ptr;
  p1.ptr = data;
  p1_pitch = p1.pitch;
  p1.pitch = p1_pitch;
  p1_x = p1.xsize;
  p1.xsize = p1_x;
  p1_y = p1.ysize;
  p1.ysize = p1_y;

  // CHECK: sycl::range<3> extent = sycl::range<3>(x, y, 1);
  hipExtent extent = make_hipExtent(x, y, 1);
  // CHECK: sycl::id<3> pos = sycl::id<3>(0, 0, 0);
  hipPos pos = make_hipPos(0, 0, 0);

  // CHECK: dpct::mem_cpy_param_wrapper p2;
  hipMemcpy3DParms p2;
  hipArray_t a1;

  // CHECK: p2.set_from_image_data(a1);
  p2.srcArray = a1;
  // CHECK: p2.set_from_pos(pos);
  p2.srcPos = pos;
  // CHECK: p2.set_to_data(p1);
  p2.dstPtr = p1;
  // CHECK: p2.set_to_pos(sycl::id<3>(0, 0, 0));
  p2.dstPos = make_hipPos(0, 0, 0);
  // CHECK: p2.set_size(extent);
  p2.extent = extent;
  // CHECK: p2.set_direction(dpct::device_to_host);
  p2.kind = hipMemcpyDeviceToHost;
  // CHECK: dpct::dpct_memcpy(&p2);
  hipMemcpy3D(&p2);

  // CHECK: dpct::pitched_data p3;
  hipPitchedPtr p3;
  
  // CHECK: p3 = dpct::dpct_malloc(sycl::range<3>(x, y, 1));
  hipMalloc3D(&p3, make_hipExtent(x, y, 1));

  // CHECK: p2.set_from_data(dpct::pitched_data(data, x, x, y));
  p2.srcPtr = make_hipPitchedPtr(data, x, x, y);
  // CHECK: p2.set_to_data(p3);
  p2.dstPtr = p3;
  // CHECK: p2.get_size()[0] = x;
  p2.extent.width = x;
  // CHECK: p2.get_size()[1] = y;
  p2.extent.height = y;
  // CHECK: p2.get_size()[2] = 1;
  p2.extent.depth = 1;
  // CHECK: p2.set_direction(dpct::host_to_device);
  p2.kind = hipMemcpyHostToDevice;
  // CHECK: dpct::dpct_memcpy(&p2);
  hipMemcpy3D(&p2);
}

