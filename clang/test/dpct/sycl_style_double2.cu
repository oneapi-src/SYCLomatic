
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --usm-level=none -out-root %T/sycl_style_double2 %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/sycl_style_double2/sycl_style_double2.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/sycl_style_double2/sycl_style_double2.dp.cpp -o %T/sycl_style_double2/sycl_style_double2.dp.o %}

// CHECK: void func3(sycl::double2 a, sycl::double2 b, sycl::double2 c) {
void func3(double2 a, double2 b, double2 c) {
}
// CHECK: void fun(sycl::double2 a) {}
void fun(double2 a) {}

// CHECK: void kernel(sycl::double2* data) {
__global__ void kernel(double2* data) {
}

// CHECK: // Removed.
static __shared__ double2 ctemp2[2]; // Removed.

// CHECK: static void gpuMain(sycl::double2 *ctemp2){
// CHECK:   int* ctempi = (int*) ctemp2;
// CHECK:   sycl::double2* ctempd =  ctemp2;
// CHECK: }
static __global__ void gpuMain(){
  int* ctempi = (int*) ctemp2;
  double2* ctempd =  ctemp2;
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.out_of_order_queue();
  // range default constructor does the right thing.
  // CHECK: sycl::double2 deflt;
  double2 deflt;

  // CHECK: sycl::double2 copyctor1 = sycl::double2(1, 2);
  double2 copyctor1 = make_double2(1, 2);

  // CHECK: sycl::double2 copyctor2 = sycl::double2(copyctor1);
  double2 copyctor2 = double2(copyctor1);

  // CHECK: sycl::double2 copyctor3(copyctor1);
  double2 copyctor3(copyctor1);

  // CHECK: func3(deflt, sycl::double2(deflt), (sycl::double2)deflt);
  func3(deflt, double2(deflt), (double2)deflt);

  // CHECK: sycl::double2 *i4;
  double2 *i4;
  // CHECK: sycl::double2 *i5;
  double2 *i5;
  // CHECK: sycl::double2 i6;
  double2 i6;
  // CHECK: sycl::double2 i7;
  double2 i7;
  // CHECK: double i = i6.x();
  double i = i6.x;
  // CHECK: i6.x() = i7.x();
  i6.x = i7.x;
  // CHECK: if (i6.x() == i7.x()) {
  if (i6.x == i7.x) {
  }
  // CHECK: sycl::double2 i2_array[10];
  double2 i2_array[10];
  // CHECK: sycl::double2 i2_array2[10];
  double2 i2_array2[10];
  // CHECK: if (i2_array[1].x() == i2_array2[1].x()) {
  if (i2_array[1].x == i2_array2[1].x) {
  }
  // CHECK: sycl::double2 x = sycl::double2(1, 2);
  double2 x = make_double2(1, 2);
  // CHECK: i4 = (sycl::double2 *)i2_array;
  i4 = (double2 *)i2_array;
  // CHECK: i7 = (sycl::double2)i6;
  i7 = (double2)i6;
  // CHECK: i7 = sycl::double2(i6);
  i7 = double2(i6);

  // CHECK: sycl::double2* data;
  // CHECK-NEXT: q_ct1.submit(
  // CHECK-NEXT:   [&](sycl::handler &cgh) {
  // CHECK-NEXT:     dpct::access_wrapper data_acc_ct0(data, cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:     cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         kernel(data_acc_ct0.get_raw_pointer());
  // CHECK-NEXT:       });
  // CHECK-NEXT:   });
  double2* data;
  kernel<<<1, 1>>>(data);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::local_accessor<sycl::double2, 1> ctemp2_acc_ct1(sycl::range<1>(2), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class gpuMain_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 64) * sycl::range<3>(1, 1, 64), sycl::range<3>(1, 1, 64)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           gpuMain(ctemp2_acc_ct1.get_multi_ptr<sycl::access::decorated::no>().get());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  gpuMain<<<64, 64>>>();
}

