// RUN: dpct --format-range=none -out-root %T/cufft-others %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/cufft-others/cufft-others.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipfft/hipfft.h>
#include <hip/hip_runtime.h>


int main() {
  //CHECK:dpct::fft::fft_engine_ptr plan;
  //CHECK-NEXT:sycl::float2* iodata;
  hipfftHandle plan;
  float2* iodata;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);

  //CHECK:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}

int foo2() {
  hipfftHandle plan_mmany64_Z2Z;
  size_t* work_size_mmany64_Z2Z;
  long long int odist_mmany64_Z2Z;
  long long int ostride_mmany64_Z2Z;
  long long int * onembed_mmany64_Z2Z;
  long long int idist_mmany64_Z2Z;
  long long int istride_mmany64_Z2Z;
  long long int* inembed_mmany64_Z2Z;
  long long int * n_mmany64_Z2Z;
  double2* odata_mmany64_Z2Z;
  double2* idata_mmany64_Z2Z;


  //CHECK:/*
  //CHECK-NEXT:DPCT1100:{{[0-9]+}}: Currently the DFT external workspace feature in the Intel(R) oneAPI Math Kernel Library (oneMKL) is only supported on GPU devices. Use the internal workspace if your code should run on non-GPU devices.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1099:{{[0-9]+}}: Verify if the default value of the direction and placement used in the function "commit" is aligning with the related "compute" function call.
  //CHECK-NEXT:*/
  //CHECK-NEXT:plan_mmany64_Z2Z->commit(&dpct::get_in_order_queue(), 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, dpct::fft::fft_type::complex_double_to_complex_double, 12, work_size_mmany64_Z2Z);
  hipfftMakePlanMany64(plan_mmany64_Z2Z, 3, n_mmany64_Z2Z, inembed_mmany64_Z2Z, istride_mmany64_Z2Z, idist_mmany64_Z2Z, onembed_mmany64_Z2Z, ostride_mmany64_Z2Z, odist_mmany64_Z2Z, HIPFFT_Z2Z, 12, work_size_mmany64_Z2Z);

  //CHECK:plan_mmany64_Z2Z->compute<sycl::double2, sycl::double2>(idata_mmany64_Z2Z, odata_mmany64_Z2Z, dpct::fft::fft_direction::forward);
  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_FORWARD);

  //CHECK:plan_mmany64_Z2Z->compute<sycl::double2, sycl::double2>(idata_mmany64_Z2Z, odata_mmany64_Z2Z, dpct::fft::fft_direction::backward);
  hipfftExecZ2Z(plan_mmany64_Z2Z, idata_mmany64_Z2Z, odata_mmany64_Z2Z, HIPFFT_BACKWARD);

  return 0;
}

int foo3(hipStream_t stream) {
  //CHECK:dpct::fft::fft_engine_ptr plan;
  //CHECK-NEXT:sycl::float2* iodata;
  hipfftHandle plan;
  float2* iodata;

  //CHECK:plan = dpct::fft::fft_engine::create(&dpct::get_in_order_queue(), 10 + 2, dpct::fft::fft_type::real_float_to_complex_float, 3);
  //CHECK-NEXT:plan->set_queue(stream);
  hipfftPlan1d(&plan, 10 + 2, HIPFFT_R2C, 3);
  hipfftSetStream(plan, stream);

  //CHECK:plan->compute<float, sycl::float2>((float*)iodata, iodata, dpct::fft::fft_direction::forward);
  hipfftExecR2C(plan, (float*)iodata, iodata);

  return 0;
}

void foo4(double x) {
  //CHECK:const int dir = -1;
  const int dir = HIPFFT_FORWARD;
  hipfftHandle plan;
  float2* iodata;
  hipfftPlan1d(&plan, 10, HIPFFT_C2C, 3);
  //CHECK:plan->compute<sycl::float2, sycl::float2>(iodata, iodata, dpct::fft::fft_direction::forward);
  //CHECK-NEXT:plan->compute<sycl::float2, sycl::float2>(iodata, iodata, dpct::fft::fft_direction::backward);
  hipfftExecC2C(plan, iodata, iodata, dir);
  hipfftExecC2C(plan, iodata, iodata, -dir);
  const double base = dir * 3.1415926 / x;
}

void foo5(double x) {
  //CHECK:int dir = -1;
  int dir = HIPFFT_FORWARD;
  hipfftHandle plan;
  float2* iodata;
  hipfftPlan1d(&plan, 10, HIPFFT_C2C, 3);
  //CHECK:plan->compute<sycl::float2, sycl::float2>(iodata, iodata, dir == 1 ? dpct::fft::fft_direction::backward : dpct::fft::fft_direction::forward);
  //CHECK-NEXT:plan->compute<sycl::float2, sycl::float2>(iodata, iodata, -dir == 1 ? dpct::fft::fft_direction::backward : dpct::fft::fft_direction::forward);
  hipfftExecC2C(plan, iodata, iodata, dir);
  hipfftExecC2C(plan, iodata, iodata, -dir);
  const double base = dir * 3.1415926 / x;
}
