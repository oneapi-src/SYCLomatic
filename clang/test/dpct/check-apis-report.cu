#include "hip/hip_runtime.h"
// UNSUPPORTED: system-windows
// RUN: dpct -report-type=apis -report-file-prefix=check_apis_report -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: cat %S/check-apis-report_csv_ref.txt > %T/check-apis-report/check_apis_report_csv_check.txt
// RUN: cat %T/check-apis-report/check_apis_report.apis.csv >>%T/check-apis-report/check_apis_report_csv_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report/check_apis_report_csv_check.txt %T/check-apis-report/check_apis_report_csv_check.txt

// RUN: dpct -output-file=output_file_all.txt -report-type=apis -report-file-prefix=stdout -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: cat %S/check-apis-report_outputfile_ref_all.txt > %T/check-apis-report/check_output_file_all.txt
// RUN: cat %T/check-apis-report/output_file_all.txt >>%T/check-apis-report/check_output_file_all.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report/check_output_file_all.txt %T/check-apis-report/check_output_file_all.txt

// RUN: dpct -report-file-prefix=report -report-type=apis  -report-format=formatted -report-only  -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: cat %S/check-apis-report_log_ref.txt > %T/check-apis-report/check_apis_report_check.txt
// RUN: cat %T/check-apis-report/report.apis.log >>%T/check-apis-report/check_apis_report_check.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report/check_apis_report_check.txt %T/check-apis-report/check_apis_report_check.txt

// RUN: dpct -output-file=output-file.txt -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: cat %S/check-apis-report_outputfile_ref.txt > %T/check-apis-report/check_output-file.txt
// RUN: cat %T/check-apis-report/output-file.txt >>%T/check-apis-report/check_output-file.txt
// RUN: FileCheck --match-full-lines --input-file %T/check-apis-report/check_output-file.txt %T/check-apis-report/check_output-file.txt

// NOMATCH-CHECK-NOT: '{{.}}'

// RUN: dpct -output-verbosity=silent  -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=NOMATCH-CHECK -allow-empty %s


// FAKE-FILE-CHECK-NOT:Processing: {{(.+)/([^/]+)}}
// FAKE-FILE-STDERR: Processing: {{(.+)/([^/]+)}}

// RUN: dpct -output-verbosity=normal  -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=FAKE-FILE-CHECK -allow-empty %s

// RUN: dpct -output-verbosity=detailed  -out-root %T/check-apis-report %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only  2>&1  \
// RUN: | FileCheck -check-prefix=FAKE-FILE-STDERR -allow-empty %s

#include <hip/hip_runtime.h>

void checkError(hipError_t err) {
}

void fooo() {
  size_t size = 10 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;

  size_t length = size * size * size;
  size_t bytes = length * sizeof(float);
  float *src;

  hipHostFree(d_A);

  hipHostMalloc(&src, bytes);

  struct hipPitchedPtr srcGPU;

  struct hipExtent extent = make_hipExtent(size * sizeof(float), size, size);

  hipMalloc3D(&srcGPU, extent);

  int2 a;
  uint4 b;
  dim3 d3;
  hipDeviceProp_t cdp;
  hipStream_t cuSt;
  const int2 c = {0,0};
  int2 d[100];
  longlong4 ll4;
}

int cool() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipMalloc((void **)&d_A, size);
  hipMemset(d_A, 0xf, size);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(h_A, d_A, size, hipMemcpyDeviceToHost);
  hipFree(d_A);
  free(h_A);
  hipDeviceSynchronize();
  hipError_t err = hipDeviceSynchronize();
  checkError(hipDeviceSynchronize());
  return hipDeviceSynchronize();
}

static texture<float, 3, hipReadModeElementType>
    NoiseTextures[4]; // texture Array

void CreateTexture();

__global__ void
AccesTexture(texture<float, 3, hipReadModeElementType> *NoiseTextures) {
  int test = tex3D(NoiseTextures[0], threadIdx.x, threadIdx.y,
                   threadIdx.z); // by using this the error occurs
}

int main(int argc, char **argv) {
  AccesTexture<<<1, dim3(4, 4, 4)>>>(NoiseTextures);
}


template<typename T>
__global__ void addKernel(T *c, const T *a, const T *b)
{
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}

#define  SIZE_X 128 //numbers in elements
#define  SIZE_Y 128
#define  SIZE_Z 128
void bar(){
  typedef float  VolumeType;
  hipExtent volumeSize = make_hipExtent(SIZE_X, SIZE_Y, SIZE_Z);
  float d_volumeMem[100];
  hipMalloc((void**)&d_volumeMem[0], SIZE_X*SIZE_Y*SIZE_Z*sizeof(float));

  hipChannelFormatDesc channelDesc = hipCreateChannelDesc<VolumeType>();
  make_hipPitchedPtr((void*)d_volumeMem, SIZE_X*sizeof(VolumeType), SIZE_X, SIZE_Y);
  hipFuncAttributes attrib;
  hipError_t err;
  err = hipFuncGetAttributes(&attrib, reinterpret_cast<const void*>(addKernel<float>));
}

namespace libsvm {
extern "C"
void SVMTrain(void){
    float* d_value_inter;
    hipDeviceptr_t base, dptr;
    size_t size_range;
    hipMemGetAddressRange(&base, &size_range, dptr);
    int a = max(1, 3);
}
}
