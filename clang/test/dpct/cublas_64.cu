// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cublas_64 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublas_64/cublas_64.dp.cpp --match-full-lines %s


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  int64_t ldc;
  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_s, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_d, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_c, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(*handle, transa, transb, m, n, k, dpct::get_value(alpha_z, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  hipblasFillMode_t uplo;
  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, uplo, transa, n, k, dpct::get_value(alpha_s, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::get_value(B_s, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, uplo, transa, n, k, dpct::get_value(alpha_d, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::get_value(B_d, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, uplo, transa, n, k, dpct::get_value(alpha_c, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(B_c, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(*handle, uplo, transa, n, k, dpct::get_value(alpha_z, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(B_z, *handle), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, C_z, ldc);
}
