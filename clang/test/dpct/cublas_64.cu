// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cublas_64 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublas_64/cublas_64.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas_64/cublas_64.dp.cpp -o %T/cublas_64/cublas_64.dp.o %}


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  float *C1_s;
  double *C1_d;
  float2 *C1_c;
  double2 *C1_z;
  int64_t ldc;
  hipblasFillMode_t uplo;
  hipblasSideMode_t side;
  hipblasDiagType_t diag;
  int64_t result;
  float result_s;
  double result_d;
  float2 result_c;
  double2 result_z;
  int64_t incx;
  int64_t incy;

  //      CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamax_64(handle, n, A_s, lda, &result);
  status = hipblasIdamax_64(handle, n, A_d, lda, &result);
  status = hipblasIcamax_64(handle, n, A_c, lda, &result);
  status = hipblasIzamax_64(handle, n, A_z, lda, &result);

  //      CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamin_64(handle, n, A_s, lda, &result);
  status = hipblasIdamin_64(handle, n, A_d, lda, &result);
  status = hipblasIcamin_64(handle, n, A_c, lda, &result);
  status = hipblasIzamin_64(handle, n, A_z, lda, &result);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSnrm2_64(handle, n, A_s, incx, &result_s);
  status = hipblasDnrm2_64(handle, n, A_d, incx, &result_d);
  status = hipblasScnrm2_64(handle, n, A_c, incx, &result_s);
  status = hipblasDznrm2_64(handle, n, A_z, incx, &result_d);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct6(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct6(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), &result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), &result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), &result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), &result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSdot_64(handle, n, A_s, incx, B_s, incy, &result_s);
  status = hipblasDdot_64(handle, n, A_d, incx, B_d, incy, &result_d);
  status = hipblasCdotu_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasCdotc_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasZdotu_64(handle, n, A_z, incx, B_z, incy, &result_z);
  status = hipblasZdotc_64(handle, n, A_z, incx, B_z, incy, &result_z);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx));
  status = hipblasSscal_64(handle, n, alpha_s, C_s, incx);
  status = hipblasDscal_64(handle, n, alpha_d, C_d, incx);
  status = hipblasCscal_64(handle, n, alpha_c, C_c, incx);
  status = hipblasZscal_64(handle, n, alpha_z, C_z, incx);
  status = hipblasCsscal_64(handle, n, alpha_s, C_c, incx);
  status = hipblasZdscal_64(handle, n, alpha_d, C_z, incx);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incy));
  status = hipblasSaxpy_64(handle, n, alpha_s, A_s, incx, C_s, incy);
  status = hipblasDaxpy_64(handle, n, alpha_d, A_d, incx, C_d, incy);
  status = hipblasCaxpy_64(handle, n, alpha_c, A_c, incx, C_c, incy);
  status = hipblasZaxpy_64(handle, n, alpha_z, A_z, incx, C_z, incy);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incy));
  status = hipblasScopy_64(handle, n, A_s, incx, C_s, incy);
  status = hipblasDcopy_64(handle, n, A_d, incx, C_d, incy);
  status = hipblasCcopy_64(handle, n, A_c, incx, C_c, incy);
  status = hipblasZcopy_64(handle, n, A_z, incx, C_z, incy);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C1_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C1_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C1_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C1_z)), incy));
  status = hipblasSswap_64(handle, n, C_s, incx, C1_s, incy);
  status = hipblasDswap_64(handle, n, C_d, incx, C1_d, incy);
  status = hipblasCswap_64(handle, n, C_c, incx, C1_c, incy);
  status = hipblasZswap_64(handle, n, C_z, incx, C1_z, incy);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_memory())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSasum_64(handle, n, A_s, incx, &result_s);
  status = hipblasDasum_64(handle, n, A_d, incx, &result_d);
  status = hipblasScasum_64(handle, n, A_c, incx, &result_s);
  status = hipblasDzasum_64(handle, n, A_z, incx, &result_d);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, beta_s, C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, beta_d, C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsymm_64(handle, side, uplo, m, n, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsymm_64(handle, side, uplo, m, n, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsymm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsymm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasStrsm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, C_s, ldc);
  status = hipblasDtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, C_d, ldc);
  status = hipblasCtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, C_c, ldc);
  status = hipblasZtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasChemm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZhemm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasCherk_64(handle, uplo, transa, n, k, alpha_s, A_c, lda, beta_s, C_c, ldc);
  status = hipblasZherk_64(handle, uplo, transa, n, k, alpha_d, A_z, lda, beta_d, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsyr2k_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsyr2k_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsyr2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsyr2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasCher2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc);
  status = hipblasZher2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSgeam_64(handle, transa, transb, m, n, alpha_s, A_s, lda, beta_s, B_s, ldb, C_s, ldc);
  status = hipblasDgeam_64(handle, transa, transb, m, n, alpha_d, A_d, lda, beta_d, B_d, ldb, C_d, ldc);
  status = hipblasCgeam_64(handle, transa, transb, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);
  status = hipblasZgeam_64(handle, transa, transb, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSdgmm_64(handle, side, m, n, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDdgmm_64(handle, side, m, n, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCdgmm_64(handle, side, m, n, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZdgmm_64(handle, side, m, n, A_z, lda, B_z, ldb, C_z, ldc);
}
