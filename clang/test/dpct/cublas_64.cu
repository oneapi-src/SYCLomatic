// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cublas_64 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublas_64/cublas_64.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas_64/cublas_64.dp.cpp -o %T/cublas_64/cublas_64.dp.o %}


#include <hip/hip_runtime.h>
#include "hipblas.h"

void foo() {
  hipblasStatus_t status;
  hipblasHandle_t handle;
  hipblasOperation_t transa;
  hipblasOperation_t transb;
  int64_t m;
  int64_t n;
  int64_t k;
  const float *alpha_s;
  const double *alpha_d;
  const float2 *alpha_c;
  const double2 *alpha_z;
  const half *alpha_h;
  const float *A_s;
  const double *A_d;
  const float2 *A_c;
  const double2 *A_z;
  const half *A_h;
  int64_t lda;
  const float *B_s;
  const double *B_d;
  const float2 *B_c;
  const double2 *B_z;
  const half *B_h;
  int64_t ldb;
  const float *beta_s;
  const double *beta_d;
  const float2 *beta_c;
  const double2 *beta_z;
  const half *beta_h;
  float *C_s;
  double *C_d;
  float2 *C_c;
  double2 *C_z;
  half *C_h;
  float *C1_s;
  double *C1_d;
  float2 *C1_c;
  double2 *C1_z;
  int64_t ldc;
  hipblasFillMode_t uplo;
  hipblasSideMode_t side;
  hipblasDiagType_t diag;
  int64_t result;
  float result_s;
  double result_d;
  float2 result_c;
  double2 result_z;
  int64_t incx;
  int64_t incy;

  int64_t elemSize;
  hipStream_t stream;
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, incy, incx, 1, n, elemSize));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, incy, incx, 1, n, elemSize));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, incy, incx, 1, n, elemSize, dpct::automatic, *stream, true));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, incy, incx, 1, n, elemSize, dpct::automatic, *stream, true));
  status = cublasSetVector_64(n, elemSize, A_s, incx, C_s, incy);
  status = cublasGetVector_64(n, elemSize, A_s, incx, C_s, incy);
  status = cublasSetVectorAsync_64(n, elemSize, A_s, incx, C_s, incy, stream);
  status = cublasGetVectorAsync_64(n, elemSize, A_s, incx, C_s, incy, stream);

  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, ldb, lda, m, n, elemSize));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, ldb, lda, m, n, elemSize));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, ldb, lda, m, n, elemSize, dpct::automatic, *stream, true));
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::matrix_mem_copy(C_s, A_s, ldb, lda, m, n, elemSize, dpct::automatic, *stream, true));
  status = cublasSetMatrix_64(m, n, elemSize, A_s, lda, C_s, ldb);
  status = cublasGetMatrix_64(m, n, elemSize, A_s, lda, C_s, ldb);
  status = cublasSetMatrixAsync_64(m, n, elemSize, A_s, lda, C_s, ldb, stream);
  status = cublasGetMatrixAsync_64(m, n, elemSize, A_s, lda, C_s, ldb, stream);

  //      CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamax_64(handle, n, A_s, lda, &result);
  status = hipblasIdamax_64(handle, n, A_d, lda, &result);
  status = hipblasIcamax_64(handle, n, A_c, lda, &result);
  status = hipblasIzamax_64(handle, n, A_z, lda, &result);

  //      CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_int64_out res_wrapper_ct4(handle->get_queue(), &result);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::int64_t>(res_wrapper_ct4.get_ptr())), oneapi::mkl::index_base::one);
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasIsamin_64(handle, n, A_s, lda, &result);
  status = hipblasIdamin_64(handle, n, A_d, lda, &result);
  status = hipblasIcamin_64(handle, n, A_c, lda, &result);
  status = hipblasIzamin_64(handle, n, A_z, lda, &result);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSnrm2_64(handle, n, A_s, incx, &result_s);
  status = hipblasDnrm2_64(handle, n, A_d, incx, &result_d);
  status = hipblasScnrm2_64(handle, n, A_c, incx, &result_s);
  status = hipblasDznrm2_64(handle, n, A_z, incx, &result_d);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct6(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct6(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), &result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), &result_c);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), &result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), &result_z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSdot_64(handle, n, A_s, incx, B_s, incy, &result_s);
  status = hipblasDdot_64(handle, n, A_d, incx, B_d, incy, &result_d);
  status = hipblasCdotu_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasCdotc_64(handle, n, A_c, incx, B_c, incy, &result_c);
  status = hipblasZdotu_64(handle, n, A_z, incx, B_z, incy, &result_z);
  status = hipblasZdotc_64(handle, n, A_z, incx, B_z, incy, &result_z);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx));
  status = hipblasSscal_64(handle, n, alpha_s, C_s, incx);
  status = hipblasDscal_64(handle, n, alpha_d, C_d, incx);
  status = hipblasCscal_64(handle, n, alpha_c, C_c, incx);
  status = hipblasZscal_64(handle, n, alpha_z, C_z, incx);
  status = hipblasCsscal_64(handle, n, alpha_s, C_c, incx);
  status = hipblasZdscal_64(handle, n, alpha_d, C_z, incx);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incy));
  status = hipblasSaxpy_64(handle, n, alpha_s, A_s, incx, C_s, incy);
  status = hipblasDaxpy_64(handle, n, alpha_d, A_d, incx, C_d, incy);
  status = hipblasCaxpy_64(handle, n, alpha_c, A_c, incx, C_c, incy);
  status = hipblasZaxpy_64(handle, n, alpha_z, A_z, incx, C_z, incy);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incy));
  status = hipblasScopy_64(handle, n, A_s, incx, C_s, incy);
  status = hipblasDcopy_64(handle, n, A_d, incx, C_d, incy);
  status = hipblasCcopy_64(handle, n, A_c, incx, C_c, incy);
  status = hipblasZcopy_64(handle, n, A_z, incx, C_z, incy);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C1_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C1_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C1_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C1_z)), incy));
  status = hipblasSswap_64(handle, n, C_s, incx, C1_s, incy);
  status = hipblasDswap_64(handle, n, C_d, incx, C1_d, incy);
  status = hipblasCswap_64(handle, n, C_c, incx, C1_c, incy);
  status = hipblasZswap_64(handle, n, C_z, incx, C1_z, incy);

  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), &result_s);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  //      CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), &result_d);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct4.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSasum_64(handle, n, A_s, incx, &result_s);
  status = hipblasDasum_64(handle, n, A_d, incx, &result_d);
  status = hipblasScasum_64(handle, n, A_c, incx, &result_s);
  status = hipblasDzasum_64(handle, n, A_z, incx, &result_d);

  const float *const_s;
  const double *const_d;
  const float2 *const_c;
  const double2 *const_z;
  float *s;
  double *d;
  float2 *c;
  double2 *z;

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C1_s)), incy, dpct::get_value(const_s, handle->get_queue()), dpct::get_value(const_s, handle->get_queue())));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C1_d)), incy, dpct::get_value(const_d, handle->get_queue()), dpct::get_value(const_d, handle->get_queue())));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C1_c)), incy, dpct::get_value(const_s, handle->get_queue()), dpct::get_value(const_c, handle->get_queue())));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C1_c)), incy, dpct::get_value(const_s, handle->get_queue()), dpct::get_value(const_s, handle->get_queue())));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C1_z)), incy, dpct::get_value(const_d, handle->get_queue()), dpct::get_value(const_z, handle->get_queue())));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rot(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C1_z)), incy, dpct::get_value(const_d, handle->get_queue()), dpct::get_value(const_d, handle->get_queue())));
  status = hipblasSrot_64(handle, n, C_s, incx, C1_s, incy, const_s, const_s);
  status = hipblasDrot_64(handle, n, C_d, incx, C1_d, incy, const_d, const_d);
  status = hipblasCrot_64(handle, n, C_c, incx, C1_c, incy, const_s, const_c);
  status = hipblasCsrot_64(handle, n, C_c, incx, C1_c, incy, const_s, const_s);
  status = hipblasZrot_64(handle, n, C_z, incx, C1_z, incy, const_d, const_z);
  status = hipblasZdrot_64(handle, n, C_z, incx, C1_z, incy, const_d, const_d);

  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_in res_wrapper_ct6(handle->get_queue(), const_s, 5);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotm(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(s)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(s)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  // CHECK: status = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_in res_wrapper_ct6(handle->get_queue(), const_d, 5);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotm(handle->get_queue(), n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d)), incx, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(d)), incy, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(res_wrapper_ct6.get_ptr())));
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  status = hipblasSrotm_64(handle, n, s, incx, s, incy, const_s);
  status = hipblasDrotm_64(handle, n, d, incx, d, incy, const_d);

  const float *x_s;
  const double *x_d;
  const float2 *x_c;
  const double2 *x_z;
  float *y_s;
  double *y_d;
  float2 *y_c;
  double2 *y_z;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), transa, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_s)), incx, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), transa, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_d)), incx, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), transa, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), transa, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasSgemv_64(handle, transa, m, n, alpha_s, A_s, lda, x_s, incx, beta_s, y_s, incy);
  status = hipblasDgemv_64(handle, transa, m, n, alpha_d, A_d, lda, x_d, incx, beta_d, y_d, incy);
  status = hipblasCgemv_64(handle, transa, m, n, alpha_c, A_c, lda, x_c, incx, beta_c, y_c, incy);
  status = hipblasZgemv_64(handle, transa, m, n, alpha_z, A_z, lda, x_z, incx, beta_z, y_z, incy);

  int64_t kl, ku;
  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), transa, m, n, kl, ku, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(x_s)), incx, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), transa, m, n, kl, ku, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(x_d)), incx, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), transa, m, n, kl, ku, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(x_c)), incx, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), transa, m, n, kl, ku, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(x_z)), incx, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasSgbmv_64(handle, transa, m, n, kl, ku, alpha_s, A_s, lda, x_s, incx, beta_s, y_s, incy);
  status = hipblasDgbmv_64(handle, transa, m, n, kl, ku, alpha_d, A_d, lda, x_d, incx, beta_d, y_d, incy);
  status = hipblasCgbmv_64(handle, transa, m, n, kl, ku, alpha_c, A_c, lda, x_c, incx, beta_c, y_c, incy);
  status = hipblasZgbmv_64(handle, transa, m, n, kl, ku, alpha_z, A_z, lda, x_z, incx, beta_z, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasStrmv_64(handle, uplo, transa, diag, n, A_s, lda, y_s, incy);
  status = hipblasDtrmv_64(handle, uplo, transa, diag, n, A_d, lda, y_d, incy);
  status = hipblasCtrmv_64(handle, uplo, transa, diag, n, A_c, lda, y_c, incy);
  status = hipblasZtrmv_64(handle, uplo, transa, diag, n, A_z, lda, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), uplo, transa, diag, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), uplo, transa, diag, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), uplo, transa, diag, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), uplo, transa, diag, n, k, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasStbmv_64(handle, uplo, transa, diag, n, k, A_s, lda, y_s, incy);
  status = hipblasDtbmv_64(handle, uplo, transa, diag, n, k, A_d, lda, y_d, incy);
  status = hipblasCtbmv_64(handle, uplo, transa, diag, n, k, A_c, lda, y_c, incy);
  status = hipblasZtbmv_64(handle, uplo, transa, diag, n, k, A_z, lda, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasStpmv_64(handle, uplo, transa, diag, n, A_s, y_s, incy);
  status = hipblasDtpmv_64(handle, uplo, transa, diag, n, A_d, y_d, incy);
  status = hipblasCtpmv_64(handle, uplo, transa, diag, n, A_c, y_c, incy);
  status = hipblasZtpmv_64(handle, uplo, transa, diag, n, A_z, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasStrsv_64(handle, uplo, transa, diag, n, A_s, lda, y_s, incy);
  status = hipblasDtrsv_64(handle, uplo, transa, diag, n, A_d, lda, y_d, incy);
  status = hipblasCtrsv_64(handle, uplo, transa, diag, n, A_c, lda, y_c, incy);
  status = hipblasZtrsv_64(handle, uplo, transa, diag, n, A_z, lda, y_z, incy);

  // CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(y_s)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(y_d)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(y_c)), incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), uplo, transa, diag, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(y_z)), incy));
  status = hipblasStpsv_64(handle, uplo, transa, diag, n, A_s, y_s, incy);
  status = hipblasDtpsv_64(handle, uplo, transa, diag, n, A_d, y_d, incy);
  status = hipblasCtpsv_64(handle, uplo, transa, diag, n, A_c, y_c, incy);
  status = hipblasZtpsv_64(handle, uplo, transa, diag, n, A_z, y_z, incy);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_h, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(A_h)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(B_h)), ldb, dpct::get_value(beta_h, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<sycl::half>(C_h)), ldc));
  status = hipblasSgemm_64(handle, transa, transb, m, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDgemm_64(handle, transa, transb, m, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCgemm_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZgemm_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);
  status = hipblasHgemm_64(handle, transa, transb, m, n, k, alpha_h, A_h, lda, B_h, ldb, beta_h, C_h, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc, oneapi::mkl::blas::compute_mode::complex_3m));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc, oneapi::mkl::blas::compute_mode::complex_3m));
  status = cublasCgemm3m_64(handle, transa, transb, m, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = cublasZgemm3m_64(handle, transa, transb, m, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT:   status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syrk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsyrk_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, beta_s, C_s, ldc);
  status = hipblasDsyrk_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, beta_d, C_d, ldc);
  status = hipblasCsyrk_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, beta_c, C_c, ldc);
  status = hipblasZsyrk_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsymm_64(handle, side, uplo, m, n, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsymm_64(handle, side, uplo, m, n, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsymm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsymm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::trsm(handle->get_queue(), side, uplo, transa, diag, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasStrsm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, C_s, ldc);
  status = hipblasDtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, C_d, ldc);
  status = hipblasCtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, C_c, ldc);
  status = hipblasZtrsm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::hemm(handle->get_queue(), side, uplo, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasChemm_64(handle, side, uplo, m, n, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZhemm_64(handle, side, uplo, m, n, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::herk(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasCherk_64(handle, uplo, transa, n, k, alpha_s, A_c, lda, beta_s, C_c, ldc);
  status = hipblasZherk_64(handle, uplo, transa, n, k, alpha_d, A_z, lda, beta_d, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSsyr2k_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsyr2k_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsyr2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsyr2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::her2k(handle->get_queue(), uplo, transa, n, k, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasCher2k_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc);
  status = hipblasZher2k_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::get_value(beta_s, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::get_value(beta_d, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::get_value(beta_c, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::get_value(beta_z, handle->get_queue()), dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSgeam_64(handle, transa, transb, m, n, alpha_s, A_s, lda, beta_s, B_s, ldb, C_s, ldc);
  status = hipblasDgeam_64(handle, transa, transb, m, n, alpha_d, A_d, lda, beta_d, B_d, ldb, C_d, ldc);
  status = hipblasCgeam_64(handle, transa, transb, m, n, alpha_c, A_c, lda, beta_c, B_c, ldb, C_c, ldc);
  status = hipblasZgeam_64(handle, transa, transb, m, n, alpha_z, A_z, lda, beta_z, B_z, ldb, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(A_s)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(B_s)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<float>(C_s)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(A_d)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(B_d)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<double>(C_d)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(A_c)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(B_c)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<float>>(C_c)), ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), side, m, n, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(A_z)), lda, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(B_z)), ldb, dpct::rvalue_ref_to_lvalue_ref(dpct::get_buffer<std::complex<double>>(C_z)), ldc));
  status = hipblasSdgmm_64(handle, side, m, n, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDdgmm_64(handle, side, m, n, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCdgmm_64(handle, side, m, n, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZdgmm_64(handle, side, m, n, A_z, lda, B_z, ldb, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::trmm(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, B_s, ldb, C_s, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::trmm(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, B_d, ldb, C_d, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::trmm(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, B_c, ldb, C_c, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::trmm(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, B_z, ldb, C_z, ldc));
  status = hipblasStrmm_64(handle, side, uplo, transa, diag, m, n, alpha_s, A_s, lda, B_s, ldb, C_s, ldc);
  status = hipblasDtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_d, A_d, lda, B_d, ldb, C_d, ldc);
  status = hipblasCtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_c, A_c, lda, B_c, ldb, C_c, ldc);
  status = hipblasZtrmm_64(handle, side, uplo, transa, diag, m, n, alpha_z, A_z, lda, B_z, ldb, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::syrk(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::syrk(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::syrk(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::syrk(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc));
  status = hipblasSsyrkx_64(handle, uplo, transa, n, k, alpha_s, A_s, lda, B_s, ldb, beta_s, C_s, ldc);
  status = hipblasDsyrkx_64(handle, uplo, transa, n, k, alpha_d, A_d, lda, B_d, ldb, beta_d, C_d, ldc);
  status = hipblasCsyrkx_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_c, C_c, ldc);
  status = hipblasZsyrkx_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_z, C_z, ldc);

  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::herk(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::herk(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc));
  status = hipblasCherkx_64(handle, uplo, transa, n, k, alpha_c, A_c, lda, B_c, ldb, beta_s, C_c, ldc);
  status = hipblasZherkx_64(handle, uplo, transa, n, k, alpha_z, A_z, lda, B_z, ldb, beta_d, C_z, ldc);

  hipDataType type_x;
  hipDataType type_y;
  hipDataType type_res;
  hipDataType type_exec;
  hipDataType type_alpha;
  hipDataType type_cs;
  void *res;
  void *x;
  void *y;
  void *alpha;
  //      CHECK: status = DPCT_CHECK_ERROR(dpct::blas::nrm2(handle, n, x, type_x, incx, res, type_res));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::dot(handle, n, x, type_x, incx, y, type_y, incy, res, type_res));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::dotc(handle, n, x, type_x, incx, y, type_y, incy, res, type_res));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::scal(handle, n, alpha, type_alpha, x, type_x, incx));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::axpy(handle, n, alpha, type_alpha, x, type_x, incx, y, type_y, incy));
  // CHECK-NEXT: status = DPCT_CHECK_ERROR(dpct::blas::rot(handle, n, x, type_x, incx, y, type_y, incy, c, s, type_cs));
  status = hipblasNrm2Ex_64(handle, n, x, type_x, incx, res, type_res, type_exec);
  status = hipblasDotEx_64(handle, n, x, type_x, incx, y, type_y, incy, res, type_res, type_exec);
  status = hipblasDotcEx_64(handle, n, x, type_x, incx, y, type_y, incy, res, type_res, type_exec);
  status = hipblasScalEx_64(handle, n, alpha, type_alpha, x, type_x, incx, type_exec);
  status = hipblasAxpyEx_64(handle, n, alpha, type_alpha, x, type_x, incx, y, type_y, incy, type_exec);
  status = hipblasRotEx_64(handle, n, x, type_x, incx, y, type_y, incy, c, s, type_cs, type_exec);
}
