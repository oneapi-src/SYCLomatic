// RUN: dpct --format-range=none --optimize-migration -out-root %T/memcpy_optimization %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/memcpy_optimization/memcpy_optimization.dp.cpp %s
#include <hip/hip_runtime.h>
#include <iostream>

int main(){

    int *dev_a, *dev_b;
    int *host_c, *host_d, *host_e, *host_f;
    bool sync;
    hipMalloc(&dev_a, sizeof(int));
    hipMalloc(&dev_a, sizeof(int));
    host_c = (int *)malloc(sizeof(int));
    host_d = (int *)malloc(sizeof(int));
    host_e = (int *)malloc(sizeof(int));
    host_f = (int *)malloc(sizeof(int));
//CHECK:  /*
//CHECK:  DPCT1114:{{[0-9]+}}: cudaMemcpy is migrated to asynchronization memcpy, assuming in the original code the source host memory is pageable memory. If the memory is not pageable, call wait() on event return by memcpy API to ensure synchronization behavior.
//CHECK:  */
//CHECK:  q_ct1.memcpy(dev_a, dev_b, sizeof(int));
//CHECK:  q_ct1.memcpy(dev_a, dev_b, sizeof(int));
    hipMemcpy(dev_a, dev_b, sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_a, dev_b, sizeof(int), hipMemcpyDeviceToDevice);

//CHECK:  if(DPCT_CHECK_ERROR(q_ct1.memcpy(dev_a, host_e, 10))) {
//CHECK:    std::cout << "failed" << std::endl;
//CHECK:  }
    if(hipMemcpy(dev_a, host_e, 10, hipMemcpyHostToDevice)) {
        std::cout << "failed" << std::endl;
    }

//CHECK:  for(int i = 0; i < 10; i++) {
//CHECK:    int src = i;
//CHECK:    q_ct1.memcpy(dev_a, &src, sizeof(int)).wait();
//CHECK:  }
    for(int i = 0; i < 10; i++) {
        int src = i;
        hipMemcpy(dev_a, &src, sizeof(int), hipMemcpyHostToDevice);
    }

//CHECK:  q_ct1.memcpy(dev_a, host_c, sizeof(int)).wait();
//CHECK:  free(host_c);
    hipMemcpy(dev_a, host_c, sizeof(int), hipMemcpyHostToDevice);
    free(host_c);
    
//CHECK:  q_ct1.memcpy(dev_a, host_d, sizeof(int));
//CHECK:  dev_ct1.queues_wait_and_throw();
//CHECK:  free(host_d);
    hipMemcpy(dev_a, host_d, sizeof(int), hipMemcpyHostToDevice);
    hipDeviceSynchronize();
    free(host_d);

//CHECK:  q_ct1.memcpy(dev_a, host_f, sizeof(int)).wait();
//CHECK:  if(sync) {
//CHECK:    dev_ct1.queues_wait_and_throw();
//CHECK:  }
    hipMemcpy(dev_a, host_f, sizeof(int), hipMemcpyHostToDevice);
    if(sync) {
      hipDeviceSynchronize();
    }
    free(host_f);
    return 0;
}