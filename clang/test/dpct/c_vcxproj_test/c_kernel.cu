// UNSUPPORTED: -linux-
// RUN: cat %S/proj_c.vcxproj > %T/proj_c.vcxproj
// RUN: cd %T

// RUN: dpct --format-range=none  --vcxprojfile=%T/proj_c.vcxproj  -in-root=%S -out-root=%T  %s %S/CuTmp_1.cu --cuda-include-path="%cuda-path/include"

// RUN: cat %S/CuTmp_1.cu > %T/CuTmp_1.dp.cpp
// RUN: cat %S/check_compilation_ref.txt  >%T/check_compilation_db.txt
// RUN: cat %T/compile_commands.json >>%T/check_compilation_db.txt
// RUN: FileCheck --match-full-lines --input-file %T/check_compilation_db.txt %T/check_compilation_db.txt
// RUN: FileCheck %S/CuTmp_1.cu --match-full-lines --input-file %T/CuTmp_1.dp.cpp

// RUN: dpct -output-file=output-file.txt --format-range=none -in-root=%S -out-root=%T/2 -p %S --process-all --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/2/c_kernel.dp.cpp --match-full-lines %S/c_kernel.cu
// RUN: FileCheck --match-full-lines --input-file %T/2/readme_3.txt %T/2/readme_3.txt

#include "hip/hip_runtime.h"
#include <stdio.h>

// CHECK: void addKernel(int *c, const int *a, const int *b, sycl::nd_item<3> item_ct1)
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // CHECK: int i = item_ct1.get_local_id(2);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
