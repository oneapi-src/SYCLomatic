// UNSUPPORTED: system-linux
// RUN: cat %S/proj_c.vcxproj > %T/proj_c.vcxproj
// RUN: cd %T

// RUN: dpct --format-range=none  --vcxprojfile=%T/proj_c.vcxproj  -in-root=%S -out-root=%T  %s %S/CuTmp_1.cu --cuda-include-path="%cuda-path/include"

// RUN: cat %S/CuTmp_1.cu > %T/CuTmp_1.dp.cpp
// RUN: cat %S/check_compilation_ref.txt  >%T/check_compilation_db.txt
// RUN: cat %T/compile_commands.json >>%T/check_compilation_db.txt
// RUN: FileCheck --match-full-lines --input-file %T/check_compilation_db.txt %T/check_compilation_db.txt
// RUN: FileCheck %S/CuTmp_1.cu --match-full-lines --input-file %T/CuTmp_1.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/CuTmp_1.dp.cpp -o %T/CuTmp_1.dp.o %}

// RUN: dpct -output-file=output-file.txt --format-range=none -in-root=%S -out-root=%T/2 -p %S --process-all --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/2/c_kernel.dp.cpp --match-full-lines %S/c_kernel.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/2/c_kernel.dp.cpp -o %T/2/c_kernel.dp.o %}
// RUN: FileCheck --match-full-lines --input-file %T/2/readme_3.txt %T/2/readme_3.txt

#include "hip/hip_runtime.h"
#include <stdio.h>

// CHECK: void addKernel(int *c, const int *a, const int *b,
// CHECK-NEXT: const sycl::nd_item<3> &item_ct1)
__global__ void addKernel(int *c, const int *a, const int *b)
{
    // CHECK: int i = item_ct1.get_local_id(2);
    int i = threadIdx.x;
    c[i] = a[i] + b[i];
}
