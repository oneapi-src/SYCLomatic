#include "hip/hip_runtime.h"
// RUN: dpct --enable-profiling  --format-range=none -out-root %T/tm-usm-restricted-profiling %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/tm-usm-restricted-profiling/tm-usm-restricted-profiling.dp.cpp --match-full-lines %s

// CHECK:#define DPCT_PROFILING_ENABLED
// CHECK-NEXT:#include <sycl/sycl.hpp>
// CHECK-NEXT:#include <dpct/dpct.hpp>
// CHECK-NEXT:#include <stdio.h>
// CHECK-NEXT:#include <cmath>
#include <stdio.h>

#define N 1000

__global__
void add(int *a, int *b) {
    int i = blockIdx.x;
    if (i<N) {
        b[i] = 2*a[i];
    }
}

int main() {
    hipStream_t stream;

    int ha[N], hb[N];
  // CHECK: dpct::event_ptr start, stop;
    hipEvent_t start, stop;
    hipError_t cudaStatus;

    int *da, *db;
    float elapsedTime;

    hipMalloc((void **)&da, N*sizeof(int));
    hipMalloc((void **)&db, N*sizeof(int));

    for (int i = 0; i<N; ++i) {
        ha[i] = i;
    }


  // CHECK:    start = new sycl::event();
  // CHECK-NEXT:    stop = new sycl::event();
    hipEventCreate(&start);
    hipEventCreate(&stop);

 // CHECK:    *start = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(start, 0);

  // CHECK: q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice);
  // CHECK: q_ct1.memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, 0);
  // CHECK: stream->memcpy(da, ha, N*sizeof(int));
    hipMemcpyAsync(da, ha, N*sizeof(int), hipMemcpyHostToDevice, stream);

  // CHECK:    *stop = q_ct1.ext_oneapi_submit_barrier();
  // CHECK-NEXT:    stop->wait_and_throw();
  // CHECK-NEXT:   elapsedTime = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventRecord(stop, 0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&elapsedTime, start, stop);

    add<<<N, 1>>>(da, db);

  // CHECK: q_ct1.memcpy(hb, db, N*sizeof(int));
    hipMemcpyAsync(hb, db, N*sizeof(int), hipMemcpyDeviceToHost);

    hipDeviceSynchronize();


    for (int i = 0; i<N; ++i) {
        printf("%d\n", hb[i]);
    }

    hipEventDestroy(start);
    hipEventDestroy(stop);

    hipFree(da);
    hipFree(db);

    return 0;
}

#define SAFE_CALL(call)                                                   \
  do {                                                                         \
    hipError_t err = call;                                                            \
  } while (0)

void foo_usm() {
  hipStream_t s1, s2;
  int *gpu_t, *host_t, n = 10;
  hipEvent_t start, stop;

// CHECK:  SAFE_CALL((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
  SAFE_CALL(hipEventRecord(start, 0));

// CHECK:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT:SAFE_CALL((s1->memcpy(gpu_t, host_t, n * sizeof(int)), 0));
  SAFE_CALL(hipMemcpyAsync(gpu_t, host_t, n * sizeof(int), hipMemcpyHostToDevice, s1));

// CHECK:  /*
// CHECK-NEXT:  DPCT1024:{{[0-9]+}}: The original code returned the error code that was further consumed by the program logic. This original code was replaced with 0. You may need to rewrite the program logic consuming the error code.
// CHECK-NEXT:  */
// CHECK-NEXT:  SAFE_CALL((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT:   SAFE_CALL((stop->wait_and_throw(), 0));
  SAFE_CALL(hipEventRecord(stop, 0));
  SAFE_CALL(hipEventSynchronize(stop));
  float Time = 0.0f;
  hipEventElapsedTime(&Time, start, stop);
}

__global__ void readTexels(int n, float *d_out, int width){}
__global__ void readTexelsFoo1(int n, float *d_out){}
__global__ void readTexelsFoo2(int n, float *d_out, int width, int height){}
texture<float4, 2, hipReadModeElementType> texA;

void foo()
{
    const unsigned int passes = 100;
    const unsigned int nsizes = 5;
    const unsigned int sizes[] = { 16, 64, 256, 1024, 4096 };
    const unsigned int kernelRepFoo[] = { 1024, 1024, 1024, 1024, 256 };
    const unsigned int iterations = 10;

    hipEvent_t start, stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);

    for (int j = 0; j < nsizes; j++)
    {
        const unsigned int size      = 1024 * sizes[j];
        const unsigned int numFloat  = size / sizeof(float);
        const unsigned int numFloat4 = size / sizeof(float4);
        size_t width, height;
        const unsigned int kernelRepFactor = kernelRepFoo[j];

        // Image memory sizes should be power of 2.
        size_t sizeLog = lround(log2(double(numFloat4)));
        height = 1 << (sizeLog >> 1);  // height is the smaller size
        width = numFloat4 / height;

        const dim3 blockSize(16, 8);
        const dim3 gridSize(width/blockSize.x, height/blockSize.y);

        float *h_in = new float[numFloat];
        float *h_out = new float[numFloat4];
        float *d_out;
        hipMalloc((void**) &d_out, numFloat4 * sizeof(float));

        // Allocate a cuda array
        hipArray* cuArray;
        hipMallocArray(&cuArray, &texA.channelDesc, width, height);

        // Copy in source data
        hipMemcpyToArray(cuArray, 0, 0, h_in, size, hipMemcpyHostToDevice);

        // Bind texture to the array
        hipBindTextureToArray(texA, cuArray);

        for (int p = 0; p < passes; p++)
        {
            // Test 1: Repeated Linear Access
            float t = 0.0f;

// CHECK:            *start = q_ct1.ext_oneapi_submit_barrier();
            hipEventRecord(start, 0);
            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {
// CHECK:                 DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                 */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexels_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexels(kernelRepFactor, d_out, width);
// CHECK-NEXT:                      });
                readTexels<<<gridSize, blockSize>>>(kernelRepFactor, d_out,
                                                    width);
            }

// CHECK:             *stop = q_ct1.ext_oneapi_submit_barrier();
// CHECK-NEXT:             stop->wait_and_throw();
// CHECK-NEXT:             t = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 2 Repeated Cache Access
// CHECK:            *start = q_ct1.ext_oneapi_submit_barrier();
            hipEventRecord(start, 0);
            for (int iter = 0; iter < iterations; iter++)
            {

// CHECK:                DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexelsFoo1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexelsFoo1(kernelRepFactor, d_out);
// CHECK-NEXT:                      });
                readTexelsFoo1<<<gridSize, blockSize>>>
                        (kernelRepFactor, d_out);
            }

// CHECK:            *stop = q_ct1.ext_oneapi_submit_barrier();
// CHECK-NEXT:            stop->wait_and_throw();
// CHECK-NEXT:            t = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

            // Verify results
            hipMemcpy(h_out, d_out, numFloat4*sizeof(float),
                    hipMemcpyDeviceToHost);

            // Test 3 Repeated "Random" Access
// CHECK:            *start = q_ct1.ext_oneapi_submit_barrier();
            hipEventRecord(start, 0);

            // read texels from texture
            for (int iter = 0; iter < iterations; iter++)
            {

// CHECK:                DPCT1049:{{[0-9]+}}: The work-group size passed to the SYCL kernel may exceed the limit. To get the device limit, query info::device::max_work_group_size. Adjust the work-group size if needed.
// CHECK-NEXT:                */
// CHECK-NEXT:                q_ct1.parallel_for<dpct_kernel_name<class readTexelsFoo2_{{[a-z0-9]+}}>>(
// CHECK-NEXT:                      sycl::nd_range<3>(gridSize * blockSize, blockSize),
// CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                        readTexelsFoo2(kernelRepFactor, d_out, width, height);
// CHECK-NEXT:                      });
                readTexelsFoo2<<<gridSize, blockSize>>>
                                (kernelRepFactor, d_out, width, height);
            }

// CHECK:            *stop = q_ct1.ext_oneapi_submit_barrier();
// CHECK-NEXT:            stop->wait_and_throw();
// CHECK-NEXT:            t = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
            hipEventRecord(stop, 0);
            hipEventSynchronize(stop);
            hipEventElapsedTime(&t, start, stop);

        }
        delete[] h_in;
        delete[] h_out;
        hipFree(d_out);
        hipFreeArray(cuArray);
        hipUnbindTexture(texA);
    }
    hipEventDestroy(start);
    hipEventDestroy(stop);
}


__global__ void kernelFunc(){}

void barr(int maxCalls) {
  hipEvent_t evtStart[maxCalls];
  hipEvent_t evtEnd[maxCalls];
  float time[maxCalls];
  for (int i = 0; i < maxCalls; i++) {
    hipEventCreate( &(evtStart[i]) );
    hipEventCreate( &(evtEnd[i]) );
    time[i] = 0.0;
  }

// CHECK: *evtStart[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[0], 0 );
  kernelFunc<<<1, 1>>>();
// CHECK:   *evtEnd[0] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[0], 0 );

// CHECK: *evtStart[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[1], 0 );

  kernelFunc<<<1, 1>>>();
// CHECK: *evtEnd[1] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[1], 0 );

// CHECK: *evtStart[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtStart[2], 0 );
  kernelFunc<<<1, 1>>>();
// CHECK: *evtEnd[2] = q_ct1.ext_oneapi_submit_barrier();
  hipEventRecord( evtEnd[2], 0 );

// CHECK: dev_ct1.queues_wait_and_throw();
  hipDeviceSynchronize();

  float total;
  int i=0;
  hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
  float timesum = 0.0f;
  for (int i = 1; i < maxCalls; i++) {
    hipEventElapsedTime( &(time[i]), evtStart[i], evtEnd[i]);
    timesum += time[i];
  }
  hipEventElapsedTime( &total, evtStart[1], evtEnd[maxCalls-1]);
}

template <class T, int blockSize>
__global__ void
reduce(const T* __restrict__ g_idata, T* __restrict__ g_odata,
       int n) {}

template <class T, class vecT>
void RunTest()
{
    int probSizes[4] = { 1, 8, 32, 64 };
    int size;
    // Convert to MiB
    size = (size * 1024 * 1024) / sizeof(T);
    // create input data on CPU
    unsigned int bytes = size * sizeof(T);

    // Allocate Host Memory
    T* h_idata;
    T* reference;
    T* h_odata;

    int num_blocks  = 64;
    int num_threads = 256;
    int smem_size = sizeof(T) * num_threads;

    // Allocate device memory
    T* d_idata, *d_odata, *d_block_sums;
    hipEvent_t start, stop;
    int passes;
    int iters;

    for (int k = 0; k < passes; k++)
    {
        float totalScanTime = 0.0f;
        SAFE_CALL(hipEventRecord(start, 0));
        for (int j = 0; j < iters; j++)
        {
// CHECK:            q_ct1.parallel_for<dpct_kernel_name<class reduce_{{[a-z0-9]+}}, T, dpct_kernel_scalar<256>>>(
// CHECK-NEXT:                  sycl::nd_range<3>(sycl::range<3>(1, 1, num_blocks) * sycl::range<3>(1, 1, num_threads), sycl::range<3>(1, 1, num_threads)),
// CHECK-NEXT:                  [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                    reduce<T, 256>(d_idata, d_block_sums, size);
// CHECK-NEXT:                  });
            reduce<T, 256><<<num_blocks, num_threads, smem_size>>>(d_idata, d_block_sums, size);
        }
        SAFE_CALL(hipEventRecord(stop, 0));
        SAFE_CALL(hipEventSynchronize(stop));
        hipEventElapsedTime(&totalScanTime, start, stop);
    }
    SAFE_CALL(hipFree(d_idata));
    SAFE_CALL(hipFree(d_odata));
    SAFE_CALL(hipFree(d_block_sums));
    SAFE_CALL(hipHostFree(h_idata));
    SAFE_CALL(hipHostFree(h_odata));
    SAFE_CALL(hipHostFree(reference));
    SAFE_CALL(hipEventDestroy(start));
    SAFE_CALL(hipEventDestroy(stop));
}

int foo_test_5() {
   RunTest<float, float4>();
}

__global__ void foo_kernel_1(unsigned short* blk_sad, unsigned short* frame,
                            int mb_width, int mb_height,
                            unsigned short* img_ref) {}

__global__ void foo_kernel_2(unsigned short* blk_sad, int mb_width,
                                  int mb_height) {}

__global__ void foo_kernel_3(unsigned short* blk_sad, int mb_width,
                                   int mb_height) {}

void test_1999(void* ref_image, void* cur_image,
                    float* sad_calc_ms, float* sad_calc_8_ms,
                    float* sad_calc_16_ms,
                    unsigned short** h_sads) {
    size_t image_width_macroblocks;
    size_t image_height_macroblocks;
    size_t image_size_macroblocks;
    size_t nsads;
    unsigned short* imgRef = NULL;
    unsigned short* d_cur_image = NULL;
    unsigned short* d_sads = NULL;

// CHECK:     dpct::event_ptr sad_calc_start, sad_calc_stop;
    hipEvent_t sad_calc_start, sad_calc_stop;
    hipEventCreate(&sad_calc_start);
    hipEventCreate(&sad_calc_stop);
// CHECK:    *sad_calc_start = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(sad_calc_start);
    dim3 foo_kernel_1_threads_in_block;
    dim3 foo_kernel_1_blocks_in_grid;

// CHECK:    q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_1_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_1_blocks_in_grid * foo_kernel_1_threads_in_block, foo_kernel_1_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_1(d_sads, d_cur_image, image_width_macroblocks, image_height_macroblocks, imgRef);
// CHECK-NEXT:          });
    foo_kernel_1<<<foo_kernel_1_blocks_in_grid,
                  foo_kernel_1_threads_in_block>>>(d_sads, d_cur_image,
                                                  image_width_macroblocks,
                                                  image_height_macroblocks,
                                                  imgRef);

// CHECK:    *sad_calc_stop = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(sad_calc_stop);

// CHECK:    dpct::event_ptr sad_calc_8_start, sad_calc_8_stop;
    hipEvent_t sad_calc_8_start, sad_calc_8_stop;

    hipEventCreate(&sad_calc_8_start);
    hipEventCreate(&sad_calc_8_stop);
// CHECK:    *sad_calc_8_start = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(sad_calc_8_start);
    dim3 foo_kernel_2_threads_in_block;
    dim3 foo_kernel_2_blocks_in_grid;

// CHECK:    q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_2_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_2_blocks_in_grid * foo_kernel_2_threads_in_block, foo_kernel_2_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_2(d_sads, image_width_macroblocks, image_height_macroblocks);
// CHECK-NEXT:          });
    foo_kernel_2<<<
      foo_kernel_2_blocks_in_grid,
      foo_kernel_2_threads_in_block>>>(d_sads, image_width_macroblocks,
                                            image_height_macroblocks);
// CHECK:    *sad_calc_8_stop = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(sad_calc_8_stop);


// CHECK:    dpct::event_ptr sad_calc_16_start, sad_calc_16_stop;
    hipEvent_t sad_calc_16_start, sad_calc_16_stop;

    hipEventCreate(&sad_calc_16_start);
    hipEventCreate(&sad_calc_16_stop);
    hipEventRecord(sad_calc_16_start);
    dim3 foo_kernel_3_threads_in_block;
    dim3 foo_kernel_3_blocks_in_grid;

// CHECK:    q_ct1.parallel_for<dpct_kernel_name<class foo_kernel_3_{{[a-z0-9]+}}>>(
// CHECK-NEXT:          sycl::nd_range<3>(foo_kernel_3_blocks_in_grid * foo_kernel_3_threads_in_block, foo_kernel_3_threads_in_block),
// CHECK-NEXT:          [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:            foo_kernel_3(d_sads, image_width_macroblocks, image_height_macroblocks);
// CHECK-NEXT:          });
    foo_kernel_3<<<
      foo_kernel_3_blocks_in_grid,
      foo_kernel_3_threads_in_block>>>(d_sads, image_width_macroblocks,
                                             image_height_macroblocks);
// CHECK:    *sad_calc_16_stop = q_ct1.ext_oneapi_submit_barrier();
    hipEventRecord(sad_calc_16_stop);

    hipHostMalloc((void **)h_sads, nsads * sizeof(unsigned short));
    hipMemcpy(*h_sads, d_sads, nsads * sizeof(*d_sads), hipMemcpyDeviceToHost);
    hipFree(d_sads);
    hipFree(d_cur_image);
    hipFree(imgRef);

// CHECK:    *(sad_calc_ms) = (sad_calc_stop->get_profiling_info<sycl::info::event_profiling::command_end>() - sad_calc_start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
// CHECK-NEXT:    *(sad_calc_8_ms) = (sad_calc_8_stop->get_profiling_info<sycl::info::event_profiling::command_end>() - sad_calc_8_start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
// CHECK-NEXT:    *(sad_calc_16_ms) = (sad_calc_16_stop->get_profiling_info<sycl::info::event_profiling::command_end>() - sad_calc_16_start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    hipEventElapsedTime(sad_calc_ms, sad_calc_start, sad_calc_stop);
    hipEventElapsedTime(sad_calc_8_ms, sad_calc_8_start, sad_calc_8_stop);
    hipEventElapsedTime(sad_calc_16_ms, sad_calc_16_start, sad_calc_16_stop);
}

__global__ void kernel() {}
void foo_test_1983() {
  hipStream_t stream1;
  hipStream_t stream2;
  hipStreamCreate(&stream1);
  hipStreamCreate(&stream2);

  hipEvent_t event1, event2;
  hipEventCreate(&event1);
  hipEventCreate(&event2);
  int repeat = 2;

  for (int i = 0; i < repeat; i++) {
    kernel<<<1, 1, 0, stream1>>>();
// CHECK:    *event1 = stream1->ext_oneapi_submit_barrier();
    hipEventRecord(event1, stream1);
    kernel<<<1, 1, 0, stream2>>>();

// CHECK:    *event2 = stream2->ext_oneapi_submit_barrier();
// CHECK-NEXT:    event1->wait_and_throw();
// CHECK-NEXT:    event2->wait_and_throw();
    hipEventRecord(event2, stream2);
    hipEventSynchronize(event1);
    hipEventSynchronize(event2);
  }
}

template <class T, class vecT> void foo_test_2131();
int foo_test_2131_host() { foo_test_2131<float, float4>(); }

template <class T, class vecT> void foo_test_2131() {
  int size;
  int num_blocks = 64;
  int num_threads = 256;
  int smem_size = sizeof(T) * num_threads;

  // Allocate device memory
  T *d_idata, *d_odata, *d_block_sums;
  hipEvent_t start, stop;
  int passes;
  int iters;

  for (int k = 0; k < passes; k++) {
    float totalScanTime = 0.0f;
  // CHECK:     SAFE_CALL((*start = q_ct1.ext_oneapi_submit_barrier(), 0));
    SAFE_CALL(hipEventRecord(start, 0));
    for (int j = 0; j < iters; j++) {
      reduce<T, 256>
          <<<num_blocks, num_threads, smem_size>>>(d_idata, d_block_sums, size);
    }

  // CHECK: SAFE_CALL((*stop = q_ct1.ext_oneapi_submit_barrier(), 0));
  // CHECK: SAFE_CALL((stop->wait_and_throw(), 0));
  // CHECK: totalScanTime = (stop->get_profiling_info<sycl::info::event_profiling::command_end>() - start->get_profiling_info<sycl::info::event_profiling::command_start>()) / 1000000.0f;
    SAFE_CALL(hipEventRecord(stop, 0));
    SAFE_CALL(hipEventSynchronize(stop));
    hipEventElapsedTime(&totalScanTime, start, stop);
  }
}
