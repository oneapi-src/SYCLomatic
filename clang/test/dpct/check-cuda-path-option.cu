// RUN: dpct --format-range=none --cuda-path="%cuda-path" -out-root %T/check-cuda-path-option %s -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/check-cuda-path-option/check-cuda-path-option.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/check-cuda-path-option/check-cuda-path-option.dp.cpp -o %T/check-cuda-path-option/check-cuda-path-option.dp.o %}

#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

// CHECK: void foo (int s){
void foo (hipblasStatus_t s){
}

