// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/cublas-usm-11 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm-11/cublas-usm-11.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas-usm-11/cublas-usm-11.dp.cpp -o %T/cublas-usm-11/cublas-usm-11.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void foo1() {
  hipblasHandle_t handle;
  void *x, *y, *a, *b, *c, *alpha, *beta, *res, *cos, *sin;
  const void **a_array;
  const void **b_array;
  void **c_array;
  //CHECK:dpct::nrm2(*handle, 4, x, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::dot(*handle, 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::dotc(*handle, 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::scal(*handle, 4, alpha, dpct::library_data_t::real_float, x, dpct::library_data_t::real_float, 1);
  //CHECK-NEXT:dpct::axpy(*handle, 4, alpha, dpct::library_data_t::real_float, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1);
  //CHECK-NEXT:dpct::rot(*handle, 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, cos, sin, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, dpct::library_data_t::real_half, 4, b, dpct::library_data_t::real_half, 4, beta, c, dpct::library_data_t::real_half, 4, dpct::library_data_t::real_half);
  //CHECK-NEXT:dpct::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a_array, dpct::library_data_t::real_half, 4, b_array, dpct::library_data_t::real_half, 4, beta, c_array, dpct::library_data_t::real_half, 4, 2, dpct::library_data_t::real_half);
  //CHECK-NEXT:dpct::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, dpct::library_data_t::real_half, 4, 16, b, dpct::library_data_t::real_half, 4, 16, beta, c, dpct::library_data_t::real_half, 4, 16, 2, dpct::library_data_t::real_half);
  hipblasNrm2Ex(handle, 4, x, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotcEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasScalEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, HIP_R_32F);
  hipblasAxpyEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, y, HIP_R_32F, 1, HIP_R_32F);
  hipblasRotEx(handle, 4, x, HIP_R_32F, 1,  y, HIP_R_32F, 1,  cos, sin, HIP_R_32F, HIP_R_32F);
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, b, HIP_R_16F, 4, beta, c, HIP_R_16F, 4, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a_array, HIP_R_16F, 4, b_array, HIP_R_16F, 4, beta, c_array, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, 16, b, HIP_R_16F, 4, 16, beta, c, HIP_R_16F, 4, 16, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
}

void foo2() {
  hipblasHandle_t handle;
  void *x, *y, *a, *b, *c, *alpha, *beta, *res, *cos, *sin;
  void **a_array;
  void **b_array;
  void **c_array;

  //CHECK:dpct::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, const_cast<void const **>(a_array), dpct::library_data_t::real_half, 4, const_cast<void const **>(b_array), dpct::library_data_t::real_half, 4, beta, c_array, dpct::library_data_t::real_half, 4, 2, dpct::library_data_t::real_half);
  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a_array, HIP_R_16F, 4, b_array, HIP_R_16F, 4, beta, c_array, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
}

void foo3() {
  hipblasHandle_t handle;
  //CHECK: int Mathmode;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetMathMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetMathMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasMath_t Mathmode;
  hipblasGetMathMode(handle, &Mathmode);
  hipblasSetMathMode(handle, Mathmode);
}
