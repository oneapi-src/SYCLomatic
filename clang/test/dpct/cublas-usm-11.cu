// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/cublas-usm-11 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm-11/cublas-usm-11.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas-usm-11/cublas-usm-11.dp.cpp -o %T/cublas-usm-11/cublas-usm-11.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

void foo1() {
  hipblasHandle_t handle;
  void *x, *y, *a, *b, *c, *alpha, *beta, *res, *cos, *sin;
  const void **a_array;
  const void **b_array;
  void **c_array;
  //CHECK:dpct::nrm2(handle->get_queue(), 4, x, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::dot(handle->get_queue(), 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::dotc(handle->get_queue(), 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, res, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::scal(handle->get_queue(), 4, alpha, dpct::library_data_t::real_float, x, dpct::library_data_t::real_float, 1);
  //CHECK-NEXT:dpct::axpy(handle->get_queue(), 4, alpha, dpct::library_data_t::real_float, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1);
  //CHECK-NEXT:dpct::rot(handle->get_queue(), 4, x, dpct::library_data_t::real_float, 1, y, dpct::library_data_t::real_float, 1, cos, sin, dpct::library_data_t::real_float);
  //CHECK-NEXT:dpct::blas::gemm(handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, dpct::library_data_t::real_half, 4, b, dpct::library_data_t::real_half, 4, beta, c, dpct::library_data_t::real_half, 4, dpct::compute_type::f16);
  //CHECK-NEXT:dpct::blas::gemm_batch(handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a_array, dpct::library_data_t::real_half, 4, b_array, dpct::library_data_t::real_half, 4, beta, c_array, dpct::library_data_t::real_half, 4, 2, dpct::compute_type::f16);
  //CHECK-NEXT:dpct::blas::gemm_batch(handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, a, dpct::library_data_t::real_half, 4, 16, b, dpct::library_data_t::real_half, 4, 16, beta, c, dpct::library_data_t::real_half, 4, 16, 2, dpct::compute_type::f16);
  hipblasNrm2Ex(handle, 4, x, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasDotcEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  hipblasScalEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, HIP_R_32F);
  hipblasAxpyEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, y, HIP_R_32F, 1, HIP_R_32F);
  hipblasRotEx(handle, 4, x, HIP_R_32F, 1,  y, HIP_R_32F, 1,  cos, sin, HIP_R_32F, HIP_R_32F);
  hipblasGemmEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, b, HIP_R_16F, 4, beta, c, HIP_R_16F, 4, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a_array, HIP_R_16F, 4, b_array, HIP_R_16F, 4, beta, c_array, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, 16, b, HIP_R_16F, 4, 16, beta, c, HIP_R_16F, 4, 16, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
}

void foo2() {
  hipblasHandle_t handle;
  void *x, *y, *a, *b, *c, *alpha, *beta, *res, *cos, *sin;
  void **a_array;
  void **b_array;
  void **c_array;

  //CHECK:dpct::blas::gemm_batch(handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 4, 4, alpha, const_cast<void const **>(a_array), dpct::library_data_t::real_half, 4, const_cast<void const **>(b_array), dpct::library_data_t::real_half, 4, beta, c_array, dpct::library_data_t::real_half, 4, 2, dpct::compute_type::f16);
  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a_array, HIP_R_16F, 4, b_array, HIP_R_16F, 4, beta, c_array, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
}

void foo3() {
  hipblasHandle_t handle;
  //CHECK: dpct::blas::math_mode Mathmode;
  //CHECK-NEXT: Mathmode = handle->get_math_mode();
  //CHECK-NEXT: handle->set_math_mode(Mathmode);
  hipblasMath_t Mathmode;
  hipblasGetMathMode(handle, &Mathmode);
  hipblasSetMathMode(handle, Mathmode);
}
