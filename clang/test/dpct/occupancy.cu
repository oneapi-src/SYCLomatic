
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/occupancy %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/occupancy/occupancy.dp.cpp

__global__ void k() {}

int main() {
  int num_blocks;
  int block_size = 128;
  size_t dynamic_shared_memory_size = 0;
  //CHECK:/*
  //CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of cudaOccupancyMaxActiveBlocksPerMultiprocessor is not supported.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cudaOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, k, block_size, dynamic_shared_memory_size);
  hipOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, k, block_size, dynamic_shared_memory_size);

  hipFunction_t func;
  //CHECK:/*
  //CHECK-NEXT:DPCT1007:{{[0-9]+}}: Migration of cuOccupancyMaxActiveBlocksPerMultiprocessor is not supported.
  //CHECK-NEXT:*/
  //CHECK-NEXT:cuOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, func, block_size, dynamic_shared_memory_size);
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(&num_blocks, func, block_size, dynamic_shared_memory_size);
  return 0;
}
