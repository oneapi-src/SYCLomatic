// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct -out-root %T/thrust_gather %s --cuda-include-path="%cuda-path/include" --usm-level=none
// RUN: FileCheck --input-file %T/thrust_gather/thrust_gather.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/thrust_gather/thrust_gather.dp.cpp -o %T/thrust_gather/thrust_gather.dp.o %}

#include <thrust/gather.h>
#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/execution_policy.h>
struct is_less_than_zero
{
   __host__ __device__
  bool operator()(int x) const
  {
    return x < 0;
  }
};

int main(void) {

  thrust::device_vector<int> AD(4);
  thrust::device_vector<int> BD(4);
  thrust::device_vector<int> SD(4);
  thrust::device_vector<int> RD(4);
  thrust::host_vector<int> AH(4);
  thrust::host_vector<int> BH(4);
  thrust::host_vector<int> SH(4);
  thrust::host_vector<int> RH(4);
  
  is_less_than_zero pred;

  int *h_ptr;
  int *d_ptr;

  h_ptr = (int*)std::malloc(20 * sizeof(int));
  hipMalloc(&d_ptr, 20 * sizeof(int));

/*******************************************************************************************
 1. Test gather_if
 2. Test four VERSIONs (with/without exec argument with/without predicate)
 3. Test each VERSION with (device_vector/host_vector/malloc-ed memory/hipMalloc-ed memory)
 *******************************************************************************************/

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::gather_if(oneapi::dpl::execution::seq, AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin(), pred);
// CHECK-NEXT:dpct::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin(), pred);
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr)) {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(SH.begin()), dpct::device_pointer<>(BH.begin()), dpct::device_pointer<>(RH.begin()), pred);
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, SH.begin(), BH.begin(), RH.begin(), pred);
// CHECK-NEXT:};
  // VERSION                        first       last      stencil     input       result      pred
  thrust::gather_if(                AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin(), pred);
  thrust::gather_if(                AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin(), pred);
  thrust::gather_if(                h_ptr,      h_ptr+4,  SH.begin(), BH.begin(), RH.begin(), pred);
  // Overload not supported with thrust
  // thrust::gather_if(                d_ptr,      d_ptr+4,  SD.begin(), BD.begin(), RD.begin(), pred);

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////
// CHECK:dpct::gather_if(oneapi::dpl::execution::seq, AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin(), pred);
// CHECK-NEXT:dpct::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin(), pred);
// CHECK-NEXT:if (dpct::is_device_ptr(h_ptr)) {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(h_ptr), dpct::device_pointer<int>(h_ptr + 4), dpct::device_pointer<>(SH.begin()), dpct::device_pointer<>(BH.begin()), dpct::device_pointer<>(RH.begin()), pred);
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, SH.begin(), BH.begin(), RH.begin(), pred);
// CHECK-NEXT:};
// CHECK-NEXT:if (dpct::is_device_ptr(d_ptr)) {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), dpct::device_pointer<int>(d_ptr), dpct::device_pointer<int>(d_ptr + 4), dpct::device_pointer<>(SD.begin()), dpct::device_pointer<>(BD.begin()), dpct::device_pointer<>(RD.begin()), pred);
// CHECK-NEXT:} else {
// CHECK-NEXT:  dpct::gather_if(oneapi::dpl::execution::seq, d_ptr, d_ptr + 4, SD.begin(), BD.begin(), RD.begin(), pred);
// CHECK-NEXT:};
  // VERSION        exec            first       last      stencil     input       result      pred
  thrust::gather_if(thrust::host,   AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin(), pred);
  thrust::gather_if(thrust::device, AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin(), pred);
  thrust::gather_if(thrust::host,   h_ptr,      h_ptr+4,  SH.begin(), BH.begin(), RH.begin(), pred);
  thrust::gather_if(thrust::device, d_ptr,      d_ptr+4,  SD.begin(), BD.begin(), RD.begin(), pred);

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// CHECK: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin());
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin());
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(h_ptr, h_ptr + 4, SH.begin(), BH.begin(), RH.begin());
  // VERSION                        first       last      stencil     input       result
  thrust::gather_if(                AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin());
  thrust::gather_if(                AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin());
  thrust::gather_if(                h_ptr,      h_ptr+4,  SH.begin(), BH.begin(), RH.begin());
  // Overload not supported with thrust
  // thrust::gather_if(                d_ptr,      d_ptr+4,  SD.begin(), BD.begin(), RD.begin());

//////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////////

// CHECK: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(oneapi::dpl::execution::seq, AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin());
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin());
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(oneapi::dpl::execution::seq, h_ptr, h_ptr + 4, SH.begin(), BH.begin(), RH.begin());
// CHECK-NEXT: /*
// CHECK-NEXT: DPCT1107:{{[0-9]+}}: Migration for this overload of thrust::gather_if is not supported.
// CHECK-NEXT: */
// CHECK-NEXT: thrust::gather_if(oneapi::dpl::execution::make_device_policy(q_ct1), d_ptr, d_ptr + 4, SD.begin(), BD.begin(), RD.begin());
  // VERSION        exec            first       last      stencil     input       result
  thrust::gather_if(thrust::host,   AH.begin(), AH.end(), SH.begin(), BH.begin(), RH.begin());
  thrust::gather_if(thrust::device, AD.begin(), AD.end(), SD.begin(), BD.begin(), RD.begin());
  thrust::gather_if(thrust::host,   h_ptr,      h_ptr+4,  SH.begin(), BH.begin(), RH.begin());
  thrust::gather_if(thrust::device, d_ptr,      d_ptr+4,  SD.begin(), BD.begin(), RD.begin());
  
  return 0;
}
