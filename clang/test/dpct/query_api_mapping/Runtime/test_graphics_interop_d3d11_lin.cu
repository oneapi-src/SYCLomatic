
#include <hip/hip_runtime.h>
// UNSUPPORTED: system-windows
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cudaGraphicsD3D11RegisterResource | FileCheck %s -check-prefix=CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE: CUDA API:
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE-NEXT:    cudaGraphicsD3D11RegisterResource(r /*cudaGraphicsResource_t **/,
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE-NEXT:                                      pD3Dr /*ID3D11Resource **/,
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE-NEXT:                                      f /*unsigned*/);
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE-NEXT: On Windows, is migrated to (with the option --use-experimental-features=bindless_images):
// CUDA_GRAPHICS_RESOURCE_D33D11_REGISTER_RESOURCE-NEXT:    r = new dpct::experimental::external_mem_wrapper(pD3Dr, f);
