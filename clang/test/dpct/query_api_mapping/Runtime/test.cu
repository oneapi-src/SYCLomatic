
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cudaStreamGetFlags | FileCheck %s -check-prefix=CUDASTREAMGETFLAGS
// CUDASTREAMGETFLAGS: CUDA API:
// CUDASTREAMGETFLAGS-NEXT:   cudaStreamGetFlags(s /*cudaStream_t*/, f /*unsigned int **/);
// CUDASTREAMGETFLAGS-NEXT: Is migrated to:
// CUDASTREAMGETFLAGS-NEXT:   *(f) = 0;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cudaEventDestroy | FileCheck %s -check-prefix=CUDAEVENTDESTROY
// CUDAEVENTDESTROY: CUDA API:
// CUDAEVENTDESTROY-NEXT:   cudaEventDestroy(e /*cudaEvent_t*/);
// CUDAEVENTDESTROY-NEXT: Is migrated to:
// CUDAEVENTDESTROY-NEXT:   dpct::destroy_event(e);
