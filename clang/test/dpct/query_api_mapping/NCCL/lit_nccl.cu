
#include <hip/hip_runtime.h>
/// Communicator Creation and Management Functions

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetLastError | FileCheck %s -check-prefix=NCCLGETLASTERROR
// NCCLGETLASTERROR: CUDA API:
// NCCLGETLASTERROR-NEXT:   ncclGetLastError(comm /*ncclComm_t*/);
// NCCLGETLASTERROR-NEXT: The API is Removed.
// NCCLGETLASTERROR-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetErrorString | FileCheck %s -check-prefix=NCCLGETERRORSTRING
// NCCLGETERRORSTRING: CUDA API:
// NCCLGETERRORSTRING-NEXT:   ncclGetErrorString(r /*ncclResult_t*/);
// NCCLGETERRORSTRING-NEXT: Is migrated to:
// NCCLGETERRORSTRING-NEXT:   dpct::get_error_dummy(r);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetVersion | FileCheck %s -check-prefix=ncclGetVersion
// ncclGetVersion: CUDA API:
// ncclGetVersion-NEXT:   ncclGetVersion(version /*int **/);
// ncclGetVersion-NEXT: Is migrated to:
// ncclGetVersion-NEXT:   *version = dpct::ccl::get_version();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclGetUniqueId | FileCheck %s -check-prefix=ncclGetUniqueId
// ncclGetUniqueId: CUDA API:
// ncclGetUniqueId-NEXT:   ncclGetUniqueId(uniqueId /*ncclUniqueId **/);
// ncclGetUniqueId-NEXT: Is migrated to:
// ncclGetUniqueId-NEXT:   *uniqueId = dpct::ccl::create_kvs_address();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommInitRank | FileCheck %s -check-prefix=ncclCommInitRank
// ncclCommInitRank: CUDA API:
// ncclCommInitRank-NEXT:   ncclCommInitRank(comm /*ncclComm_t **/, nranks /*int*/,
// ncclCommInitRank-NEXT:                   commId /*ncclUniqueId*/, rank /*int*/);
// ncclCommInitRank-NEXT: Is migrated to:
// ncclCommInitRank-NEXT:   *comm = new dpct::ccl::communicator_wrapper(nranks, rank, commId);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommDestroy | FileCheck %s -check-prefix=ncclCommDestroy
// ncclCommDestroy: CUDA API:
// ncclCommDestroy-NEXT:   ncclCommDestroy(comm /*ncclComm_t*/);
// ncclCommDestroy-NEXT: Is migrated to:
// ncclCommDestroy-NEXT:   delete comm;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommGetAsyncError | FileCheck %s -check-prefix=NCCLCOMMGETASYNCERROR
// NCCLCOMMGETASYNCERROR: CUDA API:
// NCCLCOMMGETASYNCERROR-NEXT:   ncclCommGetAsyncError(comm /*ncclComm_t*/, r /*ncclResult_t **/);
// NCCLCOMMGETASYNCERROR-NEXT: The API is Removed.
// NCCLCOMMGETASYNCERROR-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommCount | FileCheck %s -check-prefix=ncclCommCount
// ncclCommCount: CUDA API:
// ncclCommCount-NEXT:    ncclCommCount(comm /*ncclComm_t*/, count /*int **/);
// ncclCommCount-NEXT: Is migrated to:
// ncclCommCount-NEXT:   *count = comm->size();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommCuDevice | FileCheck %s -check-prefix=ncclCommCuDevice
// ncclCommCuDevice: CUDA API:
// ncclCommCuDevice-NEXT:   ncclCommCuDevice(comm /*ncclComm_t*/, device /*int **/);
// ncclCommCuDevice-NEXT: Is migrated to:
// ncclCommCuDevice-NEXT:   *device = dpct::get_device_id(comm->get_device());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclCommUserRank | FileCheck %s -check-prefix=ncclCommUserRank
// ncclCommUserRank: CUDA API:
// ncclCommUserRank-NEXT:   ncclCommUserRank(comm /*ncclComm_t*/, rank /*int **/);
// ncclCommUserRank-NEXT: Is migrated to:
// ncclCommUserRank-NEXT:   *rank = comm->rank();

/// Collective Communication Functions

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclAllReduce | FileCheck %s -check-prefix=ncclAllReduce
// ncclAllReduce: CUDA API:
// ncclAllReduce-NEXT:   ncclAllReduce(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclAllReduce-NEXT:                 datatype /*ncclDataType_t*/, op /*ncclRedOp_t*/,
// ncclAllReduce-NEXT:                 comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclAllReduce-NEXT: Is migrated to:
// ncclAllReduce-NEXT:   comm->allreduce(sendbuff, recvbuff, count, datatype, op, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclBroadcast | FileCheck %s -check-prefix=ncclBroadcast
// ncclBroadcast: CUDA API:
// ncclBroadcast-NEXT:   ncclBroadcast(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclBroadcast-NEXT:             datatype /*ncclDataType_t*/, root /*int*/, comm /*ncclComm_t*/,
// ncclBroadcast-NEXT:             stream /*cudaStream_t*/);
// ncclBroadcast-NEXT: Is migrated to:
// ncclBroadcast-NEXT:   comm->broadcast(sendbuff, recvbuff, count, datatype, root, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclBcast | FileCheck %s -check-prefix=ncclBcast
// ncclBcast: CUDA API:
// ncclBcast-NEXT:   ncclBcast(buff /*void **/, count /*size_t*/, datatype /*ncclDataType_t*/,
// ncclBcast-NEXT:                 root /*int*/, comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclBcast-NEXT: Is migrated to:
// ncclBcast-NEXT:   comm->broadcast(buff, buff, count, datatype, root, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclReduce | FileCheck %s -check-prefix=ncclReduce
// ncclReduce: CUDA API:
// ncclReduce-NEXT:   ncclReduce(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
// ncclReduce-NEXT:             datatype /*ncclDataType_t*/, op /*ncclRedOp_t*/, root /*int*/,
// ncclReduce-NEXT:             comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// ncclReduce-NEXT: Is migrated to:
// ncclReduce-NEXT:   comm->reduce(sendbuff, recvbuff, count, datatype, op, root, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclReduceScatter | FileCheck %s -check-prefix=ncclReduceScatter
// ncclReduceScatter: CUDA API:
// ncclReduceScatter-NEXT:   ncclReduceScatter(sendbuff /*void **/, recvbuff /*void **/,
// ncclReduceScatter-NEXT:                     recvcount /*size_t*/, datatype /*ncclDataType_t*/,
// ncclReduceScatter-NEXT:                     op /*ncclRedOp_t*/, comm /*ncclComm_t*/,
// ncclReduceScatter-NEXT:                     stream /*cudaStream_t*/);
// ncclReduceScatter-NEXT: Is migrated to:
// ncclReduceScatter-NEXT:   comm->reduce_scatter(sendbuff, recvbuff, recvcount, datatype, op, stream);

/// Point To Point Communication Functions

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclSend | FileCheck %s -check-prefix=NCCLSEND
// NCCLSEND: CUDA API:
// NCCLSEND-NEXT:   ncclSend(sendbuff /*const void **/, count /*size_t*/,
// NCCLSEND-NEXT:            datatype /*ncclDataType_t*/, peer /*int*/, comm /*ncclComm_t*/,
// NCCLSEND-NEXT:            stream /*cudaStream_t*/);
// NCCLSEND-NEXT: Is migrated to:
// NCCLSEND-NEXT:   comm->send(sendbuff, count, datatype, peer, stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=ncclRecv | FileCheck %s -check-prefix=NCCLRECV
// NCCLRECV: CUDA API:
// NCCLRECV-NEXT:   ncclRecv(sendbuff /*void **/, count /*size_t*/, datatype /*ncclDataType_t*/,
// NCCLRECV-NEXT:            peer /*int*/, comm /*ncclComm_t*/, stream /*cudaStream_t*/);
// NCCLRECV-NEXT: Is migrated to:
// NCCLRECV-NEXT:   comm->recv(sendbuff, count, datatype, peer, stream);
