
#include <hip/hip_runtime.h>
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsscal | FileCheck %s -check-prefix=cublasCsscal
// cublasCsscal: CUDA API:
// cublasCsscal-NEXT:   cublasCsscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const float **/,
// cublasCsscal-NEXT:                x /*cuComplex **/, incx /*int*/);
// cublasCsscal-NEXT: Is migrated to:
// cublasCsscal-NEXT:   oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIdamin | FileCheck %s -check-prefix=cublasIdamin
// cublasIdamin: CUDA API:
// cublasIdamin-NEXT:   cublasIdamin(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasIdamin-NEXT:                incx /*int*/, res /*int **/);
// cublasIdamin-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIdamin-NEXT:   [&]() {
// cublasIdamin-NEXT:   dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), res);
// cublasIdamin-NEXT:   oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, x, incx, res_wrapper_ct4.get_ptr(), oneapi::mkl::index_base::one);
// cublasIdamin-NEXT:   return 0;
// cublasIdamin-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetMatrix | FileCheck %s -check-prefix=cublasSetMatrix
// cublasSetMatrix: CUDA API:
// cublasSetMatrix-NEXT:   cublasSetMatrix(rows /*int*/, cols /*int*/, elementsize /*int*/,
// cublasSetMatrix-NEXT:                   a /*const void **/, lda /*int*/, b /*void **/, ldb /*int*/);
// cublasSetMatrix-NEXT: Is migrated to:
// cublasSetMatrix-NEXT:   dpct::matrix_mem_copy((void*)b, (void*)a, ldb, lda, rows, cols, elementsize);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZscal | FileCheck %s -check-prefix=cublasZscal
// cublasZscal: CUDA API:
// cublasZscal-NEXT:   cublasZscal(handle /*cublasHandle_t*/, n /*int*/,
// cublasZscal-NEXT:               alpha /*const cuDoubleComplex **/, x /*cuDoubleComplex **/,
// cublasZscal-NEXT:               incx /*int*/);
// cublasZscal-NEXT: Is migrated to:
// cublasZscal-NEXT:   oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrmv | FileCheck %s -check-prefix=cublasCtrmv
// cublasCtrmv: CUDA API:
// cublasCtrmv-NEXT:   cublasCtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCtrmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasCtrmv-NEXT:               n /*int*/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrmv-NEXT:               x /*cuComplex **/, incx /*int*/);
// cublasCtrmv-NEXT: Is migrated to:
// cublasCtrmv-NEXT:   oneapi::mkl::blas::column_major::trmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, (std::complex<float>*)a, lda, (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZrot | FileCheck %s -check-prefix=cublasZrot
// cublasZrot: CUDA API:
// cublasZrot-NEXT:   cublasZrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZrot-NEXT:              incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
// cublasZrot-NEXT:              c /*const double **/, s /*const cuDoubleComplex **/);
// cublasZrot-NEXT: Is migrated to:
// cublasZrot-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), n, (std::complex<double>*)x, incx, (std::complex<double>*)y, incy, dpct::get_value(c, handle->get_queue()), dpct::get_value(s, handle->get_queue()));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetVector | FileCheck %s -check-prefix=cublasSetVector
// cublasSetVector: CUDA API:
// cublasSetVector-NEXT:   cublasSetVector(n /*int*/, elementsize /*int*/, x /*const void **/,
// cublasSetVector-NEXT:                   incx /*int*/, y /*void **/, incy /*int*/);
// cublasSetVector-NEXT: Is migrated to:
// cublasSetVector-NEXT:   dpct::matrix_mem_copy((void*)y, (void*)x, incy, incx, 1, n, elementsize);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSetStream | FileCheck %s -check-prefix=cublasSetStream
// cublasSetStream: CUDA API:
// cublasSetStream-NEXT:   cublasSetStream(handle /*cublasHandle_t*/, stream /*cudaStream_t*/);
// cublasSetStream-NEXT: Is migrated to:
// cublasSetStream-NEXT:   handle->set_queue(stream);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZdgmm | FileCheck %s -check-prefix=cublasZdgmm
// cublasZdgmm: CUDA API:
// cublasZdgmm-NEXT:   cublasZdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasZdgmm-NEXT:               m /*int*/, n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZdgmm-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZdgmm-NEXT:               c /*cuDoubleComplex **/, ldc /*int*/);
// cublasZdgmm-NEXT: Is migrated to:
// cublasZdgmm-NEXT:   oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), left_right, m, n, (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZdrot | FileCheck %s -check-prefix=cublasZdrot
// cublasZdrot: CUDA API:
// cublasZdrot-NEXT:   cublasZdrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZdrot-NEXT:               incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
// cublasZdrot-NEXT:               c /*const double **/, s /*const double **/);
// cublasZdrot-NEXT: Is migrated to:
// cublasZdrot-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), n, (std::complex<double>*)x, incx, (std::complex<double>*)y, incy, dpct::get_value(c, handle->get_queue()), dpct::get_value(s, handle->get_queue()));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZhpr | FileCheck %s -check-prefix=cublasZhpr
// cublasZhpr: CUDA API:
// cublasZhpr-NEXT:   cublasZhpr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZhpr-NEXT:              n /*int*/, alpha /*const double **/, x /*const cuDoubleComplex **/,
// cublasZhpr-NEXT:              incx /*int*/, a /*cuDoubleComplex **/);
// cublasZhpr-NEXT: Is migrated to:
// cublasZhpr-NEXT:   oneapi::mkl::blas::column_major::hpr(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)x, incx, (std::complex<double>*)a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtpmv | FileCheck %s -check-prefix=cublasCtpmv
// cublasCtpmv: CUDA API:
// cublasCtpmv-NEXT:   cublasCtpmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCtpmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasCtpmv-NEXT:               n /*int*/, a /*const cuComplex **/, x /*cuComplex **/,
// cublasCtpmv-NEXT:               incx /*int*/);
// cublasCtpmv-NEXT: Is migrated to:
// cublasCtpmv-NEXT:   oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, (std::complex<float>*)a, (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDtrmv | FileCheck %s -check-prefix=cublasDtrmv
// cublasDtrmv: CUDA API:
// cublasDtrmv-NEXT:   cublasDtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDtrmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasDtrmv-NEXT:               n /*int*/, a /*const double **/, lda /*int*/, x /*double **/,
// cublasDtrmv-NEXT:               incx /*int*/);
// cublasDtrmv-NEXT: Is migrated to:
// cublasDtrmv-NEXT:   oneapi::mkl::blas::column_major::trmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrsm | FileCheck %s -check-prefix=cublasCtrsm
// cublasCtrsm: CUDA API:
// cublasCtrsm-NEXT:   cublasCtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasCtrsm-NEXT:               upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
// cublasCtrsm-NEXT:               unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
// cublasCtrsm-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrsm-NEXT:               b /*cuComplex **/, ldb /*int*/);
// cublasCtrsm-NEXT: Is migrated to:
// cublasCtrsm-NEXT:   oneapi::mkl::blas::column_major::trsm(handle->get_queue(), left_right, upper_lower, transa, unit_diag, m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, (std::complex<float>*)b, ldb);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsyrk | FileCheck %s -check-prefix=cublasCsyrk
// cublasCsyrk: CUDA API:
// cublasCsyrk-NEXT:   cublasCsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCsyrk-NEXT:               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasCsyrk-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCsyrk-NEXT:               beta /*const cuComplex **/, c /*cuComplex **/, ldc /*int*/);
// cublasCsyrk-NEXT: Is migrated to:
// cublasCsyrk-NEXT:   oneapi::mkl::blas::column_major::syrk(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDsyr2 | FileCheck %s -check-prefix=cublasDsyr2
// cublasDsyr2: CUDA API:
// cublasDsyr2-NEXT:   cublasDsyr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDsyr2-NEXT:               n /*int*/, alpha /*const double **/, x /*const double **/,
// cublasDsyr2-NEXT:               incx /*int*/, y /*const double **/, incy /*int*/, a /*double **/,
// cublasDsyr2-NEXT:               lda /*int*/);
// cublasDsyr2-NEXT: Is migrated to:
// cublasDsyr2-NEXT:   oneapi::mkl::blas::column_major::syr2(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), x, incx, y, incy, a, lda);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIdamax | FileCheck %s -check-prefix=cublasIdamax
// cublasIdamax: CUDA API:
// cublasIdamax-NEXT:   cublasIdamax(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasIdamax-NEXT:                incx /*int*/, res /*int **/);
// cublasIdamax-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIdamax-NEXT:   [&]() {
// cublasIdamax-NEXT:   dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), res);
// cublasIdamax-NEXT:   oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, x, incx, res_wrapper_ct4.get_ptr(), oneapi::mkl::index_base::one);
// cublasIdamax-NEXT:   return 0;
// cublasIdamax-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZgemv | FileCheck %s -check-prefix=cublasZgemv
// cublasZgemv: CUDA API:
// cublasZgemv-NEXT:   cublasZgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasZgemv-NEXT:               n /*int*/, alpha /*const cuDoubleComplex **/,
// cublasZgemv-NEXT:               a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZgemv-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZgemv-NEXT:               beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
// cublasZgemv-NEXT:               incy /*int*/);
// cublasZgemv-NEXT: Is migrated to:
// cublasZgemv-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans, m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasIsamin | FileCheck %s -check-prefix=cublasIsamin
// cublasIsamin: CUDA API:
// cublasIsamin-NEXT:   cublasIsamin(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
// cublasIsamin-NEXT:                incx /*int*/, res /*int **/);
// cublasIsamin-NEXT: Is migrated to (with the option --no-dry-pattern):
// cublasIsamin-NEXT:   [&]() {
// cublasIsamin-NEXT:   dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), res);
// cublasIsamin-NEXT:   oneapi::mkl::blas::column_major::iamin(handle->get_queue(), n, x, incx, res_wrapper_ct4.get_ptr(), oneapi::mkl::index_base::one);
// cublasIsamin-NEXT:   return 0;
// cublasIsamin-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZswap | FileCheck %s -check-prefix=cublasZswap
// cublasZswap: CUDA API:
// cublasZswap-NEXT:   cublasZswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
// cublasZswap-NEXT:               incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/);
// cublasZswap-NEXT: Is migrated to:
// cublasZswap-NEXT:   oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, (std::complex<double>*)x, incx, (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSspr | FileCheck %s -check-prefix=cublasSspr
// cublasSspr: CUDA API:
// cublasSspr-NEXT:   cublasSspr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasSspr-NEXT:              n /*int*/, alpha /*const float **/, x /*const float **/,
// cublasSspr-NEXT:              incx /*int*/, a /*float **/);
// cublasSspr-NEXT: Is migrated to:
// cublasSspr-NEXT:   oneapi::mkl::blas::column_major::spr(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), x, incx, a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDdot | FileCheck %s -check-prefix=cublasDdot
// cublasDdot: CUDA API:
// cublasDdot-NEXT:   cublasDdot(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasDdot-NEXT:              incx /*int*/, y /*const double **/, incy /*int*/,
// cublasDdot-NEXT:              res /*double **/);
// cublasDdot-NEXT: Is migrated to:
// cublasDdot-NEXT:   [&]() {
// cublasDdot-NEXT:   dpct::blas::wrapper_double_out res_wrapper_ct6(handle->get_queue(), res);
// cublasDdot-NEXT:   oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, x, incx, y, incy, res_wrapper_ct6.get_ptr());
// cublasDdot-NEXT:   return 0;
// cublasDdot-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCsymv | FileCheck %s -check-prefix=cublasCsymv
// cublasCsymv: CUDA API:
// cublasCsymv-NEXT:   cublasCsymv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCsymv-NEXT:               n /*int*/, alpha /*const cuComplex **/, a /*const cuComplex **/,
// cublasCsymv-NEXT:               lda /*int*/, x /*const cuComplex **/, incx /*int*/,
// cublasCsymv-NEXT:               beta /*const cuComplex **/, y /*cuComplex **/, incy /*int*/);
// cublasCsymv-NEXT: Is migrated to:
// cublasCsymv-NEXT:   oneapi::mkl::blas::column_major::symv(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, (std::complex<float>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDspr2 | FileCheck %s -check-prefix=cublasDspr2
// cublasDspr2: CUDA API:
// cublasDspr2-NEXT:   cublasDspr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDspr2-NEXT:               n /*int*/, alpha /*const double **/, x /*const double **/,
// cublasDspr2-NEXT:               incx /*int*/, y /*const double **/, incy /*int*/, a /*double **/);
// cublasDspr2-NEXT: Is migrated to:
// cublasDspr2-NEXT:   oneapi::mkl::blas::column_major::spr2(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), x, incx, y, incy, a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZherk | FileCheck %s -check-prefix=cublasZherk
// cublasZherk: CUDA API:
// cublasZherk-NEXT:   cublasZherk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZherk-NEXT:               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZherk-NEXT:               alpha /*const double **/, a /*const cuDoubleComplex **/,
// cublasZherk-NEXT:               lda /*int*/, beta /*const double **/, c /*cuDoubleComplex **/,
// cublasZherk-NEXT:               ldc /*int*/);
// cublasZherk-NEXT: Is migrated to:
// cublasZherk-NEXT:   oneapi::mkl::blas::column_major::herk(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)c, ldc);
