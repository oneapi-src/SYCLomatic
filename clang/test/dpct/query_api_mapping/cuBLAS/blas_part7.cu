
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZher2k | FileCheck %s -check-prefix=cublasZher2k
// cublasZher2k: CUDA API:
// cublasZher2k-NEXT:   cublasZher2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZher2k-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZher2k-NEXT:                alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
// cublasZher2k-NEXT:                lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
// cublasZher2k-NEXT:                beta /*const double **/, c /*cuDoubleComplex **/, ldc /*int*/);
// cublasZher2k-NEXT: Is migrated to:
// cublasZher2k-NEXT:   oneapi::mkl::blas::column_major::her2k(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)b, ldb, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDsyr2k | FileCheck %s -check-prefix=cublasDsyr2k
// cublasDsyr2k: CUDA API:
// cublasDsyr2k-NEXT:   cublasDsyr2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDsyr2k-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasDsyr2k-NEXT:                alpha /*const double **/, a /*const double **/, lda /*int*/,
// cublasDsyr2k-NEXT:                b /*const double **/, ldb /*int*/, beta /*const double **/,
// cublasDsyr2k-NEXT:                c /*double **/, ldc /*int*/);
// cublasDsyr2k-NEXT: Is migrated to:
// cublasDsyr2k-NEXT:   oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDgemm | FileCheck %s -check-prefix=cublasDgemm
// cublasDgemm: CUDA API:
// cublasDgemm-NEXT:   cublasDgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasDgemm-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasDgemm-NEXT:               alpha /*const double **/, a /*const double **/, lda /*int*/,
// cublasDgemm-NEXT:               b /*const double **/, ldb /*int*/, beta /*const double **/,
// cublasDgemm-NEXT:               c /*double **/, ldc /*int*/);
// cublasDgemm-NEXT: Is migrated to:
// cublasDgemm-NEXT:   oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasGetPointerMode | FileCheck %s -check-prefix=cublasGetPointerMode
// cublasGetPointerMode: CUDA API:
// cublasGetPointerMode-NEXT:   cublasGetPointerMode(handle /*cublasHandle_t*/,
// cublasGetPointerMode-NEXT:                        host_device /*cublasPointerMode_t **/);
// cublasGetPointerMode-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSrotg | FileCheck %s -check-prefix=cublasSrotg
// cublasSrotg: CUDA API:
// cublasSrotg-NEXT:   cublasSrotg(handle /*cublasHandle_t*/, a /*float **/, b /*float **/,
// cublasSrotg-NEXT:               c /*float **/, s /*float **/);
// cublasSrotg-NEXT: Is migrated to:
// cublasSrotg-NEXT:   [&]() {
// cublasSrotg-NEXT:   dpct::blas::wrapper_float_inout res_wrapper_ct1(handle->get_queue(), a);
// cublasSrotg-NEXT:   dpct::blas::wrapper_float_inout res_wrapper_ct2(handle->get_queue(), b);
// cublasSrotg-NEXT:   dpct::blas::wrapper_float_out res_wrapper_ct3(handle->get_queue(), c);
// cublasSrotg-NEXT:   dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), s);
// cublasSrotg-NEXT:   oneapi::mkl::blas::column_major::rotg(handle->get_queue(), res_wrapper_ct1.get_ptr(), res_wrapper_ct2.get_ptr(), res_wrapper_ct3.get_ptr(), res_wrapper_ct4.get_ptr());
// cublasSrotg-NEXT:   return 0;
// cublasSrotg-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSgemmEx | FileCheck %s -check-prefix=cublasSgemmEx
// cublasSgemmEx: CUDA API:
// cublasSgemmEx-NEXT:   cublasSgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasSgemmEx-NEXT:                 transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasSgemmEx-NEXT:                 alpha /*const float **/, a /*const void **/,
// cublasSgemmEx-NEXT:                 atype /*cudaDataType*/, lda /*int*/, b /*const void **/,
// cublasSgemmEx-NEXT:                 btype /*cudaDataType*/, ldb /*int*/, beta /*const float **/,
// cublasSgemmEx-NEXT:                 c /*void **/, ctype /*cudaDataType*/, ldc /*int*/);
// cublasSgemmEx-NEXT: Is migrated to:
// cublasSgemmEx-NEXT:   dpct::blas::gemm(handle, transa, transb, m, n, k, alpha, a, atype, lda, b, btype, ldb, beta, c, ctype, ldc, dpct::library_data_t::real_float);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDrotmg | FileCheck %s -check-prefix=cublasDrotmg
// cublasDrotmg: CUDA API:
// cublasDrotmg-NEXT:   cublasDrotmg(handle /*cublasHandle_t*/, d1 /*double **/, d2 /*double **/,
// cublasDrotmg-NEXT:                x1 /*double **/, y1 /*const double **/, param /*double **/);
// cublasDrotmg-NEXT: Is migrated to:
// cublasDrotmg-NEXT:   [&]() {
// cublasDrotmg-NEXT:   dpct::blas::wrapper_double_inout res_wrapper_ct1(handle->get_queue(), d1);
// cublasDrotmg-NEXT:   dpct::blas::wrapper_double_inout res_wrapper_ct2(handle->get_queue(), d2);
// cublasDrotmg-NEXT:   dpct::blas::wrapper_double_inout res_wrapper_ct3(handle->get_queue(), x1);
// cublasDrotmg-NEXT:   dpct::blas::wrapper_double_out res_wrapper_ct5(handle->get_queue(), param, 5);
// cublasDrotmg-NEXT:   oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), res_wrapper_ct1.get_ptr(), res_wrapper_ct2.get_ptr(), res_wrapper_ct3.get_ptr(), dpct::get_value(y1, handle->get_queue()), res_wrapper_ct5.get_ptr());
// cublasDrotmg-NEXT:   return 0;
// cublasDrotmg-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStpmv | FileCheck %s -check-prefix=cublasStpmv
// cublasStpmv: CUDA API:
// cublasStpmv-NEXT:   cublasStpmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStpmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStpmv-NEXT:               n /*int*/, a /*const float **/, x /*float **/, incx /*int*/);
// cublasStpmv-NEXT: Is migrated to:
// cublasStpmv-NEXT:   oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStrsv | FileCheck %s -check-prefix=cublasStrsv
// cublasStrsv: CUDA API:
// cublasStrsv-NEXT:   cublasStrsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStrsv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStrsv-NEXT:               n /*int*/, a /*const float **/, lda /*int*/, x /*float **/,
// cublasStrsv-NEXT:               incx /*int*/);
// cublasStrsv-NEXT: Is migrated to:
// cublasStrsv-NEXT:   oneapi::mkl::blas::column_major::trsv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStbmv | FileCheck %s -check-prefix=cublasStbmv
// cublasStbmv: CUDA API:
// cublasStbmv-NEXT:   cublasStbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStbmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStbmv-NEXT:               n /*int*/, k /*int*/, a /*const float **/, lda /*int*/,
// cublasStbmv-NEXT:               x /*float **/, incx /*int*/);
// cublasStbmv-NEXT: Is migrated to:
// cublasStbmv-NEXT:   oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, k, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStpsv | FileCheck %s -check-prefix=cublasStpsv
// cublasStpsv: CUDA API:
// cublasStpsv-NEXT:   cublasStpsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStpsv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStpsv-NEXT:               n /*int*/, a /*const float **/, x /*float **/, incx /*int*/);
// cublasStpsv-NEXT: Is migrated to:
// cublasStpsv-NEXT:   oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCherkx | FileCheck %s -check-prefix=cublasCherkx
// cublasCherkx: CUDA API:
// cublasCherkx-NEXT:   cublasCherkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCherkx-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasCherkx-NEXT:                alpha /*const cuComplex **/, a /*const cuComplex **/,
// cublasCherkx-NEXT:                lda /*int*/, b /*const cuComplex **/, ldb /*int*/,
// cublasCherkx-NEXT:                beta /*const float **/, c /*cuComplex **/, ldc /*int*/);
// cublasCherkx-NEXT: Is migrated to:
// cublasCherkx-NEXT:   dpct::blas::herk(handle, upper_lower, trans, n, k, alpha, a, lda, b, ldb, beta, c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCgeam | FileCheck %s -check-prefix=cublasCgeam
// cublasCgeam: CUDA API:
// cublasCgeam-NEXT:   cublasCgeam(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasCgeam-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/,
// cublasCgeam-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCgeam-NEXT:               beta /*const cuComplex **/, b /*const cuComplex **/, ldb /*int*/,
// cublasCgeam-NEXT:               c /*cuComplex **/, ldc /*int*/);
// cublasCgeam-NEXT: Is migrated to:
// cublasCgeam-NEXT:   oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)b, ldb, (std::complex<float>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZrotg | FileCheck %s -check-prefix=cublasZrotg
// cublasZrotg: CUDA API:
// cublasZrotg-NEXT:   cublasZrotg(handle /*cublasHandle_t*/, a /*cuDoubleComplex **/,
// cublasZrotg-NEXT:               b /*cuDoubleComplex **/, c /*double **/, s /*cuDoubleComplex **/);
// cublasZrotg-NEXT: Is migrated to:
// cublasZrotg-NEXT:   [&]() {
// cublasZrotg-NEXT:   dpct::blas::wrapper_double2_inout res_wrapper_ct1(handle->get_queue(), a);
// cublasZrotg-NEXT:   dpct::blas::wrapper_double2_inout res_wrapper_ct2(handle->get_queue(), b);
// cublasZrotg-NEXT:   dpct::blas::wrapper_double_out res_wrapper_ct3(handle->get_queue(), c);
// cublasZrotg-NEXT:   dpct::blas::wrapper_double2_out res_wrapper_ct4(handle->get_queue(), s);
// cublasZrotg-NEXT:   oneapi::mkl::blas::column_major::rotg(handle->get_queue(), (std::complex<double>*)res_wrapper_ct1.get_ptr(), (std::complex<double>*)res_wrapper_ct2.get_ptr(), res_wrapper_ct3.get_ptr(), (std::complex<double>*)res_wrapper_ct4.get_ptr());
// cublasZrotg-NEXT:   return 0;
// cublasZrotg-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasHgemm | FileCheck %s -check-prefix=cublasHgemm
// cublasHgemm: CUDA API:
// cublasHgemm-NEXT:   cublasHgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasHgemm-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasHgemm-NEXT:               alpha /*const __half **/, a /*const __half **/, lda /*int*/,
// cublasHgemm-NEXT:               b /*const __half **/, ldb /*int*/, beta /*const __half **/,
// cublasHgemm-NEXT:               c /*__half **/, ldc /*int*/);
// cublasHgemm-NEXT: Is migrated to:
// cublasHgemm-NEXT:   oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSgemv | FileCheck %s -check-prefix=cublasSgemv
// cublasSgemv: CUDA API:
// cublasSgemv-NEXT:   cublasSgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasSgemv-NEXT:               n /*int*/, alpha /*const float **/, a /*const float **/,
// cublasSgemv-NEXT:               lda /*int*/, x /*const float **/, incx /*int*/,
// cublasSgemv-NEXT:               beta /*const float **/, y /*float **/, incy /*int*/);
// cublasSgemv-NEXT: Is migrated to:
// cublasSgemv-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans, m, n, dpct::get_value(alpha, handle->get_queue()), a, lda, x, incx, dpct::get_value(beta, handle->get_queue()), y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSrotmg | FileCheck %s -check-prefix=cublasSrotmg
// cublasSrotmg: CUDA API:
// cublasSrotmg-NEXT:   cublasSrotmg(handle /*cublasHandle_t*/, d1 /*float **/, d2 /*float **/,
// cublasSrotmg-NEXT:                x1 /*float **/, y1 /*const float **/, param /*float **/);
// cublasSrotmg-NEXT: Is migrated to:
// cublasSrotmg-NEXT:   [&]() {
// cublasSrotmg-NEXT:   dpct::blas::wrapper_float_inout res_wrapper_ct1(handle->get_queue(), d1);
// cublasSrotmg-NEXT:   dpct::blas::wrapper_float_inout res_wrapper_ct2(handle->get_queue(), d2);
// cublasSrotmg-NEXT:   dpct::blas::wrapper_float_inout res_wrapper_ct3(handle->get_queue(), x1);
// cublasSrotmg-NEXT:   dpct::blas::wrapper_float_out res_wrapper_ct5(handle->get_queue(), param, 5);
// cublasSrotmg-NEXT:   oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), res_wrapper_ct1.get_ptr(), res_wrapper_ct2.get_ptr(), res_wrapper_ct3.get_ptr(), dpct::get_value(y1, handle->get_queue()), res_wrapper_ct5.get_ptr());
// cublasSrotmg-NEXT:   return 0;
// cublasSrotmg-NEXT:   }();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZhbmv | FileCheck %s -check-prefix=cublasZhbmv
// cublasZhbmv: CUDA API:
// cublasZhbmv-NEXT:   cublasZhbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZhbmv-NEXT:               n /*int*/, k /*int*/, alpha /*const cuDoubleComplex **/,
// cublasZhbmv-NEXT:               a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZhbmv-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZhbmv-NEXT:               beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
// cublasZhbmv-NEXT:               incy /*int*/);
// cublasZhbmv-NEXT: Is migrated to:
// cublasZhbmv-NEXT:   oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), upper_lower, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCswap | FileCheck %s -check-prefix=cublasCswap
// cublasCswap: CUDA API:
// cublasCswap-NEXT:   cublasCswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuComplex **/,
// cublasCswap-NEXT:               incx /*int*/, y /*cuComplex **/, incy /*int*/);
// cublasCswap-NEXT: Is migrated to:
// cublasCswap-NEXT:   oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, (std::complex<float>*)x, incx, (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCher2 | FileCheck %s -check-prefix=cublasCher2
// cublasCher2: CUDA API:
// cublasCher2-NEXT:   cublasCher2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCher2-NEXT:               n /*int*/, alpha /*const cuComplex **/, x /*const cuComplex **/,
// cublasCher2-NEXT:               incx /*int*/, y /*const cuComplex **/, incy /*int*/,
// cublasCher2-NEXT:               a /*cuComplex **/, lda /*int*/);
// cublasCher2-NEXT: Is migrated to:
// cublasCher2-NEXT:   oneapi::mkl::blas::column_major::her2(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx, (std::complex<float>*)y, incy, (std::complex<float>*)a, lda);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCaxpy | FileCheck %s -check-prefix=cublasCaxpy
// cublasCaxpy: CUDA API:
// cublasCaxpy-NEXT:   cublasCaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const cuComplex **/,
// cublasCaxpy-NEXT:               x /*const cuComplex **/, incx /*int*/, y /*cuComplex **/,
// cublasCaxpy-NEXT:               incy /*int*/);
// cublasCaxpy-NEXT: Is migrated to:
// cublasCaxpy-NEXT:   oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx, (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDaxpy | FileCheck %s -check-prefix=cublasDaxpy
// cublasDaxpy: CUDA API:
// cublasDaxpy-NEXT:   cublasDaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
// cublasDaxpy-NEXT:               x /*const double **/, incx /*int*/, y /*double **/, incy /*int*/);
// cublasDaxpy-NEXT: Is migrated to:
// cublasDaxpy-NEXT:   oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), x, incx, y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDscal | FileCheck %s -check-prefix=cublasDscal
// cublasDscal: CUDA API:
// cublasDscal-NEXT:   cublasDscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
// cublasDscal-NEXT:               x /*double **/, incx /*int*/);
// cublasDscal-NEXT: Is migrated to:
// cublasDscal-NEXT:   oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrmm | FileCheck %s -check-prefix=cublasCtrmm
// cublasCtrmm: CUDA API:
// cublasCtrmm-NEXT:   cublasCtrmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasCtrmm-NEXT:               upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
// cublasCtrmm-NEXT:               unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
// cublasCtrmm-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrmm-NEXT:               b /*const cuComplex **/, ldb /*int*/, c /*cuComplex **/,
// cublasCtrmm-NEXT:               ldc /*int*/);
// cublasCtrmm-NEXT: Is migrated to:
// cublasCtrmm-NEXT:   dpct::blas::trmm(handle, left_right, upper_lower, transa, unit_diag, m, n, alpha, a, lda, b, ldb, c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCgbmv | FileCheck %s -check-prefix=cublasCgbmv
// cublasCgbmv: CUDA API:
// cublasCgbmv-NEXT:   cublasCgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasCgbmv-NEXT:               n /*int*/, kl /*int*/, ku /*int*/, alpha /*const cuComplex **/,
// cublasCgbmv-NEXT:               a /*const cuComplex **/, lda /*int*/, x /*const cuComplex **/,
// cublasCgbmv-NEXT:               incx /*int*/, beta /*const cuComplex **/, y /*cuComplex **/,
// cublasCgbmv-NEXT:               incy /*int*/);
// cublasCgbmv-NEXT: Is migrated to:
// cublasCgbmv-NEXT:   oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), trans, m, n, kl, ku, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, (std::complex<float>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)y, incy);
