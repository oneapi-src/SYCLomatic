
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZher2k | FileCheck %s -check-prefix=cublasZher2k
// cublasZher2k: CUDA API:
// cublasZher2k-NEXT:   cublasZher2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZher2k-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZher2k-NEXT:                alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
// cublasZher2k-NEXT:                lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
// cublasZher2k-NEXT:                beta /*const double **/, c /*cuDoubleComplex **/, ldc /*int*/);
// cublasZher2k-NEXT: Is migrated to:
// cublasZher2k-NEXT:   oneapi::mkl::blas::column_major::her2k(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)b, ldb, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDsyr2k | FileCheck %s -check-prefix=cublasDsyr2k
// cublasDsyr2k: CUDA API:
// cublasDsyr2k-NEXT:   cublasDsyr2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDsyr2k-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasDsyr2k-NEXT:                alpha /*const double **/, a /*const double **/, lda /*int*/,
// cublasDsyr2k-NEXT:                b /*const double **/, ldb /*int*/, beta /*const double **/,
// cublasDsyr2k-NEXT:                c /*double **/, ldc /*int*/);
// cublasDsyr2k-NEXT: Is migrated to:
// cublasDsyr2k-NEXT:   oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDgemm | FileCheck %s -check-prefix=cublasDgemm
// cublasDgemm: CUDA API:
// cublasDgemm-NEXT:   cublasDgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasDgemm-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasDgemm-NEXT:               alpha /*const double **/, a /*const double **/, lda /*int*/,
// cublasDgemm-NEXT:               b /*const double **/, ldb /*int*/, beta /*const double **/,
// cublasDgemm-NEXT:               c /*double **/, ldc /*int*/);
// cublasDgemm-NEXT: Is migrated to:
// cublasDgemm-NEXT:   oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasGetPointerMode | FileCheck %s -check-prefix=cublasGetPointerMode
// cublasGetPointerMode: CUDA API:
// cublasGetPointerMode-NEXT:   cublasGetPointerMode(handle /*cublasHandle_t*/,
// cublasGetPointerMode-NEXT:                        host_device /*cublasPointerMode_t **/);
// cublasGetPointerMode-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSrotg | FileCheck %s -check-prefix=cublasSrotg
// cublasSrotg: CUDA API:
// cublasSrotg-NEXT:   cublasSrotg(handle /*cublasHandle_t*/, a /*float **/, b /*float **/,
// cublasSrotg-NEXT:               c /*float **/, s /*float **/);
// cublasSrotg-NEXT: Is migrated to:
// cublasSrotg-NEXT:   float* a_ct{{[0-9]+}} = a;
// cublasSrotg-NEXT:   float* b_ct{{[0-9]+}} = b;
// cublasSrotg-NEXT:   float* c_ct{{[0-9]+}} = c;
// cublasSrotg-NEXT:   float* s_ct{{[0-9]+}} = s;
// cublasSrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSrotg-NEXT:     a_ct{{[0-9]+}} = sycl::malloc_shared<float>(4, dpct::get_default_queue());
// cublasSrotg-NEXT:     b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
// cublasSrotg-NEXT:     c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
// cublasSrotg-NEXT:     s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
// cublasSrotg-NEXT:     *a_ct{{[0-9]+}} = *a;
// cublasSrotg-NEXT:     *b_ct{{[0-9]+}} = *b;
// cublasSrotg-NEXT:     *c_ct{{[0-9]+}} = *c;
// cublasSrotg-NEXT:     *s_ct{{[0-9]+}} = *s;
// cublasSrotg-NEXT:   }
// cublasSrotg-NEXT:   oneapi::mkl::blas::column_major::rotg(handle->get_queue(), a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}});
// cublasSrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSrotg-NEXT:     handle->get_queue().wait();
// cublasSrotg-NEXT:     *a = *a_ct{{[0-9]+}};
// cublasSrotg-NEXT:     *b = *b_ct{{[0-9]+}};
// cublasSrotg-NEXT:     *c = *c_ct{{[0-9]+}};
// cublasSrotg-NEXT:     *s = *s_ct{{[0-9]+}};
// cublasSrotg-NEXT:     sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasSrotg-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSgemmEx | FileCheck %s -check-prefix=cublasSgemmEx
// cublasSgemmEx: CUDA API:
// cublasSgemmEx-NEXT:   cublasSgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasSgemmEx-NEXT:                 transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasSgemmEx-NEXT:                 alpha /*const float **/, a /*const void **/,
// cublasSgemmEx-NEXT:                 atype /*cudaDataType*/, lda /*int*/, b /*const void **/,
// cublasSgemmEx-NEXT:                 btype /*cudaDataType*/, ldb /*int*/, beta /*const float **/,
// cublasSgemmEx-NEXT:                 c /*void **/, ctype /*cudaDataType*/, ldc /*int*/);
// cublasSgemmEx-NEXT: Is migrated to:
// cublasSgemmEx-NEXT:   dpct::gemm(handle->get_queue(), transa, transb, m, n, k, alpha, a, atype, lda, b, btype, ldb, beta, c, ctype, ldc, dpct::library_data_t::real_float);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDrotmg | FileCheck %s -check-prefix=cublasDrotmg
// cublasDrotmg: CUDA API:
// cublasDrotmg-NEXT:   cublasDrotmg(handle /*cublasHandle_t*/, d1 /*double **/, d2 /*double **/,
// cublasDrotmg-NEXT:                x1 /*double **/, y1 /*const double **/, param /*double **/);
// cublasDrotmg-NEXT: Is migrated to:
// cublasDrotmg-NEXT:   double* d1_ct{{[0-9]+}} = d1;
// cublasDrotmg-NEXT:   double* d2_ct{{[0-9]+}} = d2;
// cublasDrotmg-NEXT:   double* x1_ct{{[0-9]+}} = x1;
// cublasDrotmg-NEXT:   double* param_ct{{[0-9]+}} = param;
// cublasDrotmg-NEXT:   if(sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasDrotmg-NEXT:     d1_ct{{[0-9]+}} = sycl::malloc_shared<double>(8, dpct::get_default_queue());
// cublasDrotmg-NEXT:     d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
// cublasDrotmg-NEXT:     x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
// cublasDrotmg-NEXT:     param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
// cublasDrotmg-NEXT:     *d1_ct{{[0-9]+}} = *d1;
// cublasDrotmg-NEXT:     *d2_ct{{[0-9]+}} = *d2;
// cublasDrotmg-NEXT:     *x1_ct{{[0-9]+}} = *x1;
// cublasDrotmg-NEXT:   }
// cublasDrotmg-NEXT:   oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1, handle->get_queue()), param_ct{{[0-9]+}});
// cublasDrotmg-NEXT:   if(sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasDrotmg-NEXT:     handle->get_queue().wait();
// cublasDrotmg-NEXT:     *d1 = *d1_ct{{[0-9]+}};
// cublasDrotmg-NEXT:     *d2 = *d2_ct{{[0-9]+}};
// cublasDrotmg-NEXT:     *x1 = *x1_ct{{[0-9]+}};
// cublasDrotmg-NEXT:     dpct::get_default_queue().memcpy(param, param_ct{{[0-9]+}}, sizeof(double)*5).wait();
// cublasDrotmg-NEXT:     sycl::free(d1_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasDrotmg-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStpmv | FileCheck %s -check-prefix=cublasStpmv
// cublasStpmv: CUDA API:
// cublasStpmv-NEXT:   cublasStpmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStpmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStpmv-NEXT:               n /*int*/, a /*const float **/, x /*float **/, incx /*int*/);
// cublasStpmv-NEXT: Is migrated to:
// cublasStpmv-NEXT:   oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStrsv | FileCheck %s -check-prefix=cublasStrsv
// cublasStrsv: CUDA API:
// cublasStrsv-NEXT:   cublasStrsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStrsv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStrsv-NEXT:               n /*int*/, a /*const float **/, lda /*int*/, x /*float **/,
// cublasStrsv-NEXT:               incx /*int*/);
// cublasStrsv-NEXT: Is migrated to:
// cublasStrsv-NEXT:   oneapi::mkl::blas::column_major::trsv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStbmv | FileCheck %s -check-prefix=cublasStbmv
// cublasStbmv: CUDA API:
// cublasStbmv-NEXT:   cublasStbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStbmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStbmv-NEXT:               n /*int*/, k /*int*/, a /*const float **/, lda /*int*/,
// cublasStbmv-NEXT:               x /*float **/, incx /*int*/);
// cublasStbmv-NEXT: Is migrated to:
// cublasStbmv-NEXT:   oneapi::mkl::blas::column_major::tbmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, k, a, lda, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasStpsv | FileCheck %s -check-prefix=cublasStpsv
// cublasStpsv: CUDA API:
// cublasStpsv-NEXT:   cublasStpsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasStpsv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasStpsv-NEXT:               n /*int*/, a /*const float **/, x /*float **/, incx /*int*/);
// cublasStpsv-NEXT: Is migrated to:
// cublasStpsv-NEXT:   oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCherkx | FileCheck %s -check-prefix=cublasCherkx
// cublasCherkx: CUDA API:
// cublasCherkx-NEXT:   cublasCherkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCherkx-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasCherkx-NEXT:                alpha /*const cuComplex **/, a /*const cuComplex **/,
// cublasCherkx-NEXT:                lda /*int*/, b /*const cuComplex **/, ldb /*int*/,
// cublasCherkx-NEXT:                beta /*const float **/, c /*cuComplex **/, ldc /*int*/);
// cublasCherkx-NEXT: Is migrated to:
// cublasCherkx-NEXT:   dpct::herk(handle->get_queue(), upper_lower, trans, n, k, alpha, a, lda, b, ldb, beta, c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCgeam | FileCheck %s -check-prefix=cublasCgeam
// cublasCgeam: CUDA API:
// cublasCgeam-NEXT:   cublasCgeam(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasCgeam-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/,
// cublasCgeam-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCgeam-NEXT:               beta /*const cuComplex **/, b /*const cuComplex **/, ldb /*int*/,
// cublasCgeam-NEXT:               c /*cuComplex **/, ldc /*int*/);
// cublasCgeam-NEXT: Is migrated to:
// cublasCgeam-NEXT:   oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), transa, transb, m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)b, ldb, (std::complex<float>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZrotg | FileCheck %s -check-prefix=cublasZrotg
// cublasZrotg: CUDA API:
// cublasZrotg-NEXT:   cublasZrotg(handle /*cublasHandle_t*/, a /*cuDoubleComplex **/,
// cublasZrotg-NEXT:               b /*cuDoubleComplex **/, c /*double **/, s /*cuDoubleComplex **/);
// cublasZrotg-NEXT: Is migrated to:
// cublasZrotg-NEXT:   sycl::double2* a_ct{{[0-9]+}} = a;
// cublasZrotg-NEXT:   sycl::double2* b_ct{{[0-9]+}} = b;
// cublasZrotg-NEXT:   double* c_ct{{[0-9]+}} = c;
// cublasZrotg-NEXT:   sycl::double2* s_ct{{[0-9]+}} = s;
// cublasZrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasZrotg-NEXT:     a_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(3, dpct::get_default_queue());
// cublasZrotg-NEXT:     c_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
// cublasZrotg-NEXT:     b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
// cublasZrotg-NEXT:     s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
// cublasZrotg-NEXT:     *a_ct{{[0-9]+}} = *a;
// cublasZrotg-NEXT:     *b_ct{{[0-9]+}} = *b;
// cublasZrotg-NEXT:     *c_ct{{[0-9]+}} = *c;
// cublasZrotg-NEXT:     *s_ct{{[0-9]+}} = *s;
// cublasZrotg-NEXT:   }
// cublasZrotg-NEXT:   oneapi::mkl::blas::column_major::rotg(handle->get_queue(), (std::complex<double>*)a_ct{{[0-9]+}}, (std::complex<double>*)b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, (std::complex<double>*)s_ct{{[0-9]+}});
// cublasZrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasZrotg-NEXT:     handle->get_queue().wait();
// cublasZrotg-NEXT:     *a = *a_ct{{[0-9]+}};
// cublasZrotg-NEXT:     *b = *b_ct{{[0-9]+}};
// cublasZrotg-NEXT:     *c = *c_ct{{[0-9]+}};
// cublasZrotg-NEXT:     *s = *s_ct{{[0-9]+}};
// cublasZrotg-NEXT:     sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasZrotg-NEXT:     sycl::free(c_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasZrotg-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasHgemm | FileCheck %s -check-prefix=cublasHgemm
// cublasHgemm: CUDA API:
// cublasHgemm-NEXT:   cublasHgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasHgemm-NEXT:               transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasHgemm-NEXT:               alpha /*const __half **/, a /*const __half **/, lda /*int*/,
// cublasHgemm-NEXT:               b /*const __half **/, ldb /*int*/, beta /*const __half **/,
// cublasHgemm-NEXT:               c /*__half **/, ldc /*int*/);
// cublasHgemm-NEXT: Is migrated to:
// cublasHgemm-NEXT:   oneapi::mkl::blas::column_major::gemm(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb, dpct::get_value(beta, handle->get_queue()), c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSgemv | FileCheck %s -check-prefix=cublasSgemv
// cublasSgemv: CUDA API:
// cublasSgemv-NEXT:   cublasSgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasSgemv-NEXT:               n /*int*/, alpha /*const float **/, a /*const float **/,
// cublasSgemv-NEXT:               lda /*int*/, x /*const float **/, incx /*int*/,
// cublasSgemv-NEXT:               beta /*const float **/, y /*float **/, incy /*int*/);
// cublasSgemv-NEXT: Is migrated to:
// cublasSgemv-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans, m, n, dpct::get_value(alpha, handle->get_queue()), a, lda, x, incx, dpct::get_value(beta, handle->get_queue()), y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSrotmg | FileCheck %s -check-prefix=cublasSrotmg
// cublasSrotmg: CUDA API:
// cublasSrotmg-NEXT:   cublasSrotmg(handle /*cublasHandle_t*/, d1 /*float **/, d2 /*float **/,
// cublasSrotmg-NEXT:                x1 /*float **/, y1 /*const float **/, param /*float **/);
// cublasSrotmg-NEXT: Is migrated to:
// cublasSrotmg-NEXT:   float* d1_ct{{[0-9]+}} = d1;
// cublasSrotmg-NEXT:   float* d2_ct{{[0-9]+}} = d2;
// cublasSrotmg-NEXT:   float* x1_ct{{[0-9]+}} = x1;
// cublasSrotmg-NEXT:   float* param_ct{{[0-9]+}} = param;
// cublasSrotmg-NEXT:   if(sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSrotmg-NEXT:     d1_ct{{[0-9]+}} = sycl::malloc_shared<float>(8, dpct::get_default_queue());
// cublasSrotmg-NEXT:     d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
// cublasSrotmg-NEXT:     x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
// cublasSrotmg-NEXT:     param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
// cublasSrotmg-NEXT:     *d1_ct{{[0-9]+}} = *d1;
// cublasSrotmg-NEXT:     *d2_ct{{[0-9]+}} = *d2;
// cublasSrotmg-NEXT:     *x1_ct{{[0-9]+}} = *x1;
// cublasSrotmg-NEXT:   }
// cublasSrotmg-NEXT:   oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1, handle->get_queue()), param_ct{{[0-9]+}});
// cublasSrotmg-NEXT:   if(sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(d1, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSrotmg-NEXT:     handle->get_queue().wait();
// cublasSrotmg-NEXT:     *d1 = *d1_ct{{[0-9]+}};
// cublasSrotmg-NEXT:     *d2 = *d2_ct{{[0-9]+}};
// cublasSrotmg-NEXT:     *x1 = *x1_ct{{[0-9]+}};
// cublasSrotmg-NEXT:     dpct::get_default_queue().memcpy(param, param_ct{{[0-9]+}}, sizeof(float)*5).wait();
// cublasSrotmg-NEXT:     sycl::free(d1_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasSrotmg-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZhbmv | FileCheck %s -check-prefix=cublasZhbmv
// cublasZhbmv: CUDA API:
// cublasZhbmv-NEXT:   cublasZhbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZhbmv-NEXT:               n /*int*/, k /*int*/, alpha /*const cuDoubleComplex **/,
// cublasZhbmv-NEXT:               a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZhbmv-NEXT:               x /*const cuDoubleComplex **/, incx /*int*/,
// cublasZhbmv-NEXT:               beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
// cublasZhbmv-NEXT:               incy /*int*/);
// cublasZhbmv-NEXT: Is migrated to:
// cublasZhbmv-NEXT:   oneapi::mkl::blas::column_major::hbmv(handle->get_queue(), upper_lower, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCswap | FileCheck %s -check-prefix=cublasCswap
// cublasCswap: CUDA API:
// cublasCswap-NEXT:   cublasCswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuComplex **/,
// cublasCswap-NEXT:               incx /*int*/, y /*cuComplex **/, incy /*int*/);
// cublasCswap-NEXT: Is migrated to:
// cublasCswap-NEXT:   oneapi::mkl::blas::column_major::swap(handle->get_queue(), n, (std::complex<float>*)x, incx, (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCher2 | FileCheck %s -check-prefix=cublasCher2
// cublasCher2: CUDA API:
// cublasCher2-NEXT:   cublasCher2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCher2-NEXT:               n /*int*/, alpha /*const cuComplex **/, x /*const cuComplex **/,
// cublasCher2-NEXT:               incx /*int*/, y /*const cuComplex **/, incy /*int*/,
// cublasCher2-NEXT:               a /*cuComplex **/, lda /*int*/);
// cublasCher2-NEXT: Is migrated to:
// cublasCher2-NEXT:   oneapi::mkl::blas::column_major::her2(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx, (std::complex<float>*)y, incy, (std::complex<float>*)a, lda);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCaxpy | FileCheck %s -check-prefix=cublasCaxpy
// cublasCaxpy: CUDA API:
// cublasCaxpy-NEXT:   cublasCaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const cuComplex **/,
// cublasCaxpy-NEXT:               x /*const cuComplex **/, incx /*int*/, y /*cuComplex **/,
// cublasCaxpy-NEXT:               incy /*int*/);
// cublasCaxpy-NEXT: Is migrated to:
// cublasCaxpy-NEXT:   oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx, (std::complex<float>*)y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDaxpy | FileCheck %s -check-prefix=cublasDaxpy
// cublasDaxpy: CUDA API:
// cublasDaxpy-NEXT:   cublasDaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
// cublasDaxpy-NEXT:               x /*const double **/, incx /*int*/, y /*double **/, incy /*int*/);
// cublasDaxpy-NEXT: Is migrated to:
// cublasDaxpy-NEXT:   oneapi::mkl::blas::column_major::axpy(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), x, incx, y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDscal | FileCheck %s -check-prefix=cublasDscal
// cublasDscal: CUDA API:
// cublasDscal-NEXT:   cublasDscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
// cublasDscal-NEXT:               x /*double **/, incx /*int*/);
// cublasDscal-NEXT: Is migrated to:
// cublasDscal-NEXT:   oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtrmm | FileCheck %s -check-prefix=cublasCtrmm
// cublasCtrmm: CUDA API:
// cublasCtrmm-NEXT:   cublasCtrmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasCtrmm-NEXT:               upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
// cublasCtrmm-NEXT:               unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
// cublasCtrmm-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCtrmm-NEXT:               b /*const cuComplex **/, ldb /*int*/, c /*cuComplex **/,
// cublasCtrmm-NEXT:               ldc /*int*/);
// cublasCtrmm-NEXT: Is migrated to:
// cublasCtrmm-NEXT:   dpct::trmm(handle->get_queue(), left_right, upper_lower, transa, unit_diag, m, n, alpha, a, lda, b, ldb, c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCgbmv | FileCheck %s -check-prefix=cublasCgbmv
// cublasCgbmv: CUDA API:
// cublasCgbmv-NEXT:   cublasCgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasCgbmv-NEXT:               n /*int*/, kl /*int*/, ku /*int*/, alpha /*const cuComplex **/,
// cublasCgbmv-NEXT:               a /*const cuComplex **/, lda /*int*/, x /*const cuComplex **/,
// cublasCgbmv-NEXT:               incx /*int*/, beta /*const cuComplex **/, y /*cuComplex **/,
// cublasCgbmv-NEXT:               incy /*int*/);
// cublasCgbmv-NEXT: Is migrated to:
// cublasCgbmv-NEXT:   oneapi::mkl::blas::column_major::gbmv(handle->get_queue(), trans, m, n, kl, ku, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, (std::complex<float>*)x, incx, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)y, incy);
