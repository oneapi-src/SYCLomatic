
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZdscal | FileCheck %s -check-prefix=cublasZdscal
// cublasZdscal: CUDA API:
// cublasZdscal-NEXT:   cublasZdscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
// cublasZdscal-NEXT:                x /*cuDoubleComplex **/, incx /*int*/);
// cublasZdscal-NEXT: Is migrated to:
// cublasZdscal-NEXT:   oneapi::mkl::blas::column_major::scal(handle->get_queue(), n, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasGetVectorAsync | FileCheck %s -check-prefix=cublasGetVectorAsync
// cublasGetVectorAsync: CUDA API:
// cublasGetVectorAsync-NEXT:   cublasGetVectorAsync(n /*int*/, elementsize /*int*/, from /*const void **/,
// cublasGetVectorAsync-NEXT:                        incx /*int*/, to /*void **/, incy /*int*/,
// cublasGetVectorAsync-NEXT:                        stream /*cudaStream_t*/);
// cublasGetVectorAsync-NEXT: Is migrated to:
// cublasGetVectorAsync-NEXT:   dpct::matrix_mem_copy((void*)to, (void*)from, incy, incx, 1, n, elementsize, dpct::automatic, *stream, true);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZsyrk | FileCheck %s -check-prefix=cublasZsyrk
// cublasZsyrk: CUDA API:
// cublasZsyrk-NEXT:   cublasZsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZsyrk-NEXT:               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZsyrk-NEXT:               alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
// cublasZsyrk-NEXT:               lda /*int*/, beta /*const cuDoubleComplex **/,
// cublasZsyrk-NEXT:               c /*cuDoubleComplex **/, ldc /*int*/);
// cublasZsyrk-NEXT: Is migrated to:
// cublasZsyrk-NEXT:   oneapi::mkl::blas::column_major::syrk(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDtrsm | FileCheck %s -check-prefix=cublasDtrsm
// cublasDtrsm: CUDA API:
// cublasDtrsm-NEXT:   cublasDtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasDtrsm-NEXT:               upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
// cublasDtrsm-NEXT:               unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
// cublasDtrsm-NEXT:               alpha /*const double **/, a /*const double **/, lda /*int*/,
// cublasDtrsm-NEXT:               b /*double **/, ldb /*int*/);
// cublasDtrsm-NEXT: Is migrated to:
// cublasDtrsm-NEXT:   oneapi::mkl::blas::column_major::trsm(handle->get_queue(), left_right, upper_lower, transa, unit_diag, m, n, dpct::get_value(alpha, handle->get_queue()), a, lda, b, ldb);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCgemmStridedBatched | FileCheck %s -check-prefix=cublasCgemmStridedBatched
// cublasCgemmStridedBatched: CUDA API:
// cublasCgemmStridedBatched-NEXT:   cublasCgemmStridedBatched(
// cublasCgemmStridedBatched-NEXT:       handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
// cublasCgemmStridedBatched-NEXT:       transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
// cublasCgemmStridedBatched-NEXT:       alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasCgemmStridedBatched-NEXT:       stridea /*long long int*/, b /*const cuComplex **/, ldb /*int*/,
// cublasCgemmStridedBatched-NEXT:       strideb /*long long int*/, beta /*const cuComplex **/, c /*cuComplex **/,
// cublasCgemmStridedBatched-NEXT:       ldc /*int*/, stridec /*long long int*/, group_count /*int*/);
// cublasCgemmStridedBatched-NEXT: Is migrated to:
// cublasCgemmStridedBatched-NEXT:   oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, stridea, (std::complex<float>*)b, ldb, strideb, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)c, ldc, stridec, group_count);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasChemm | FileCheck %s -check-prefix=cublasChemm
// cublasChemm: CUDA API:
// cublasChemm-NEXT:   cublasChemm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasChemm-NEXT:               upper_lower /*cublasFillMode_t*/, m /*int*/, n /*int*/,
// cublasChemm-NEXT:               alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
// cublasChemm-NEXT:               b /*const cuComplex **/, ldb /*int*/, beta /*const cuComplex **/,
// cublasChemm-NEXT:               c /*cuComplex **/, ldc /*int*/);
// cublasChemm-NEXT: Is migrated to:
// cublasChemm-NEXT:   oneapi::mkl::blas::column_major::hemm(handle->get_queue(), left_right, upper_lower, m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)a, lda, (std::complex<float>*)b, ldb, dpct::get_value(beta, handle->get_queue()), (std::complex<float>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDrotg | FileCheck %s -check-prefix=cublasDrotg
// cublasDrotg: CUDA API:
// cublasDrotg-NEXT:   cublasDrotg(handle /*cublasHandle_t*/, a /*double **/, b /*double **/,
// cublasDrotg-NEXT:               c /*double **/, s /*double **/);
// cublasDrotg-NEXT: Is migrated to:
// cublasDrotg-NEXT:   double* a_ct{{[0-9]+}} = a;
// cublasDrotg-NEXT:   double* b_ct{{[0-9]+}} = b;
// cublasDrotg-NEXT:   double* c_ct{{[0-9]+}} = c;
// cublasDrotg-NEXT:   double* s_ct{{[0-9]+}} = s;
// cublasDrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasDrotg-NEXT:     a_ct{{[0-9]+}} = sycl::malloc_shared<double>(4, dpct::get_default_queue());
// cublasDrotg-NEXT:     b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
// cublasDrotg-NEXT:     c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
// cublasDrotg-NEXT:     s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
// cublasDrotg-NEXT:     *a_ct{{[0-9]+}} = *a;
// cublasDrotg-NEXT:     *b_ct{{[0-9]+}} = *b;
// cublasDrotg-NEXT:     *c_ct{{[0-9]+}} = *c;
// cublasDrotg-NEXT:     *s_ct{{[0-9]+}} = *s;
// cublasDrotg-NEXT:   }
// cublasDrotg-NEXT:   oneapi::mkl::blas::column_major::rotg(handle->get_queue(), a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}});
// cublasDrotg-NEXT:   if(sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasDrotg-NEXT:     handle->get_queue().wait();
// cublasDrotg-NEXT:     *a = *a_ct{{[0-9]+}};
// cublasDrotg-NEXT:     *b = *b_ct{{[0-9]+}};
// cublasDrotg-NEXT:     *c = *c_ct{{[0-9]+}};
// cublasDrotg-NEXT:     *s = *s_ct{{[0-9]+}};
// cublasDrotg-NEXT:     sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasDrotg-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCtpsv | FileCheck %s -check-prefix=cublasCtpsv
// cublasCtpsv: CUDA API:
// cublasCtpsv-NEXT:   cublasCtpsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasCtpsv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasCtpsv-NEXT:               n /*int*/, a /*const cuComplex **/, x /*cuComplex **/,
// cublasCtpsv-NEXT:               incx /*int*/);
// cublasCtpsv-NEXT: Is migrated to:
// cublasCtpsv-NEXT:   oneapi::mkl::blas::column_major::tpsv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, (std::complex<float>*)a, (std::complex<float>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasAxpyEx | FileCheck %s -check-prefix=cublasAxpyEx
// cublasAxpyEx: CUDA API:
// cublasAxpyEx-NEXT:   cublasAxpyEx(handle /*cublasHandle_t*/, n /*int*/, alpha /*const void **/,
// cublasAxpyEx-NEXT:                alphatype /*cudaDataType*/, x /*const void **/,
// cublasAxpyEx-NEXT:                xtype /*cudaDataType*/, incx /*int*/, y /*void **/,
// cublasAxpyEx-NEXT:                ytype /*cudaDataType*/, incy /*int*/,
// cublasAxpyEx-NEXT:                computetype /*cudaDataType*/);
// cublasAxpyEx-NEXT: Is migrated to:
// cublasAxpyEx-NEXT:   dpct::axpy(handle->get_queue(), n, alpha, alphatype, x, xtype, incx, y, ytype, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDtpmv | FileCheck %s -check-prefix=cublasDtpmv
// cublasDtpmv: CUDA API:
// cublasDtpmv-NEXT:   cublasDtpmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDtpmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasDtpmv-NEXT:               n /*int*/, a /*const double **/, x /*double **/, incx /*int*/);
// cublasDtpmv-NEXT: Is migrated to:
// cublasDtpmv-NEXT:   oneapi::mkl::blas::column_major::tpmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, a, x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZtrmv | FileCheck %s -check-prefix=cublasZtrmv
// cublasZtrmv: CUDA API:
// cublasZtrmv-NEXT:   cublasZtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZtrmv-NEXT:               trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
// cublasZtrmv-NEXT:               n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
// cublasZtrmv-NEXT:               x /*cuDoubleComplex **/, incx /*int*/);
// cublasZtrmv-NEXT: Is migrated to:
// cublasZtrmv-NEXT:   oneapi::mkl::blas::column_major::trmv(handle->get_queue(), upper_lower, trans, unit_nonunit, n, (std::complex<double>*)a, lda, (std::complex<double>*)x, incx);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSdot | FileCheck %s -check-prefix=cublasSdot
// cublasSdot: CUDA API:
// cublasSdot-NEXT:   cublasSdot(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
// cublasSdot-NEXT:              incx /*int*/, y /*const float **/, incy /*int*/, res /*float **/);
// cublasSdot-NEXT: Is migrated to:
// cublasSdot-NEXT:   float* res_temp_ptr_ct{{[0-9]+}} = res;
// cublasSdot-NEXT:   if(sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSdot-NEXT:     res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
// cublasSdot-NEXT:   }
// cublasSdot-NEXT:   oneapi::mkl::blas::column_major::dot(handle->get_queue(), n, x, incx, y, incy, res_temp_ptr_ct{{[0-9]+}});
// cublasSdot-NEXT:   if(sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasSdot-NEXT:     handle->get_queue().wait();
// cublasSdot-NEXT:     *res = *res_temp_ptr_ct{{[0-9]+}};
// cublasSdot-NEXT:     sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasSdot-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasGetVector | FileCheck %s -check-prefix=cublasGetVector
// cublasGetVector: CUDA API:
// cublasGetVector-NEXT:   cublasGetVector(n /*int*/, elementsize /*int*/, x /*const void **/,
// cublasGetVector-NEXT:                   incx /*int*/, y /*void **/, incy /*int*/);
// cublasGetVector-NEXT: Is migrated to:
// cublasGetVector-NEXT:   dpct::matrix_mem_copy((void*)y, (void*)x, incy, incx, 1, n, elementsize);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDgemv | FileCheck %s -check-prefix=cublasDgemv
// cublasDgemv: CUDA API:
// cublasDgemv-NEXT:   cublasDgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
// cublasDgemv-NEXT:               n /*int*/, alpha /*const double **/, a /*const double **/,
// cublasDgemv-NEXT:               lda /*int*/, x /*const double **/, incx /*int*/,
// cublasDgemv-NEXT:               beta /*const double **/, y /*double **/, incy /*int*/);
// cublasDgemv-NEXT: Is migrated to:
// cublasDgemv-NEXT:   oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans, m, n, dpct::get_value(alpha, handle->get_queue()), a, lda, x, incx, dpct::get_value(beta, handle->get_queue()), y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasChpr | FileCheck %s -check-prefix=cublasChpr
// cublasChpr: CUDA API:
// cublasChpr-NEXT:   cublasChpr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasChpr-NEXT:              n /*int*/, alpha /*const float **/, x /*const cuComplex **/,
// cublasChpr-NEXT:              incx /*int*/, a /*cuComplex **/);
// cublasChpr-NEXT: Is migrated to:
// cublasChpr-NEXT:   oneapi::mkl::blas::column_major::hpr(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<float>*)x, incx, (std::complex<float>*)a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZgeru | FileCheck %s -check-prefix=cublasZgeru
// cublasZgeru: CUDA API:
// cublasZgeru-NEXT:   cublasZgeru(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
// cublasZgeru-NEXT:               alpha /*const cuDoubleComplex **/, x /*const cuDoubleComplex **/,
// cublasZgeru-NEXT:               incx /*int*/, y /*const cuDoubleComplex **/, incy /*int*/,
// cublasZgeru-NEXT:               a /*cuDoubleComplex **/, lda /*int*/);
// cublasZgeru-NEXT: Is migrated to:
// cublasZgeru-NEXT:   oneapi::mkl::blas::column_major::geru(handle->get_queue(), m, n, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)x, incx, (std::complex<double>*)y, incy, (std::complex<double>*)a, lda);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCdgmm | FileCheck %s -check-prefix=cublasCdgmm
// cublasCdgmm: CUDA API:
// cublasCdgmm-NEXT:   cublasCdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
// cublasCdgmm-NEXT:               m /*int*/, n /*int*/, a /*const cuComplex **/, lda /*int*/,
// cublasCdgmm-NEXT:               x /*const cuComplex **/, incx /*int*/, c /*cuComplex **/,
// cublasCdgmm-NEXT:               ldc /*int*/);
// cublasCdgmm-NEXT: Is migrated to:
// cublasCdgmm-NEXT:   oneapi::mkl::blas::column_major::dgmm_batch(handle->get_queue(), left_right, m, n, (std::complex<float>*)a, lda, 0, (std::complex<float>*)x, incx, 0, (std::complex<float>*)c, ldc, ldc * n, 1);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDcopy | FileCheck %s -check-prefix=cublasDcopy
// cublasDcopy: CUDA API:
// cublasDcopy-NEXT:   cublasDcopy(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
// cublasDcopy-NEXT:               incx /*int*/, y /*double **/, incy /*int*/);
// cublasDcopy-NEXT: Is migrated to:
// cublasDcopy-NEXT:   oneapi::mkl::blas::column_major::copy(handle->get_queue(), n, x, incx, y, incy);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasSsyrkx | FileCheck %s -check-prefix=cublasSsyrkx
// cublasSsyrkx: CUDA API:
// cublasSsyrkx-NEXT:   cublasSsyrkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasSsyrkx-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasSsyrkx-NEXT:                alpha /*const float **/, a /*const float **/, lda /*int*/,
// cublasSsyrkx-NEXT:                b /*const float **/, ldb /*int*/, beta /*const float **/,
// cublasSsyrkx-NEXT:                c /*float **/, ldc /*int*/);
// cublasSsyrkx-NEXT: Is migrated to:
// cublasSsyrkx-NEXT:   dpct::syrk(handle->get_queue(), upper_lower, trans, n, k, alpha, a, lda, b, ldb, beta, c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDestroy | FileCheck %s -check-prefix=cublasDestroy
// cublasDestroy: CUDA API:
// cublasDestroy-NEXT:   cublasDestroy(handle /*cublasHandle_t*/);
// cublasDestroy-NEXT: Is migrated to:
// cublasDestroy-NEXT:   handle = nullptr;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasGetAtomicsMode | FileCheck %s -check-prefix=cublasGetAtomicsMode
// cublasGetAtomicsMode: CUDA API:
// cublasGetAtomicsMode-NEXT:   cublasGetAtomicsMode(handle /*cublasHandle_t*/,
// cublasGetAtomicsMode-NEXT:                        atomics /*cublasAtomicsMode_t **/);
// cublasGetAtomicsMode-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasZsyr2k | FileCheck %s -check-prefix=cublasZsyr2k
// cublasZsyr2k: CUDA API:
// cublasZsyr2k-NEXT:   cublasZsyr2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasZsyr2k-NEXT:                trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
// cublasZsyr2k-NEXT:                alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
// cublasZsyr2k-NEXT:                lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
// cublasZsyr2k-NEXT:                beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
// cublasZsyr2k-NEXT:                ldc /*int*/);
// cublasZsyr2k-NEXT: Is migrated to:
// cublasZsyr2k-NEXT:   oneapi::mkl::blas::column_major::syr2k(handle->get_queue(), upper_lower, trans, n, k, dpct::get_value(alpha, handle->get_queue()), (std::complex<double>*)a, lda, (std::complex<double>*)b, ldb, dpct::get_value(beta, handle->get_queue()), (std::complex<double>*)c, ldc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDrotm | FileCheck %s -check-prefix=cublasDrotm
// cublasDrotm: CUDA API:
// cublasDrotm-NEXT:   cublasDrotm(handle /*cublasHandle_t*/, n /*int*/, x /*double **/,
// cublasDrotm-NEXT:               incx /*int*/, y /*double **/, incy /*int*/,
// cublasDrotm-NEXT:               param /*const double **/);
// cublasDrotm-NEXT: Is migrated to:
// cublasDrotm-NEXT:   oneapi::mkl::blas::column_major::rotm(handle->get_queue(), n, x, incx, y, incy, const_cast<double*>(param));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasCdotu | FileCheck %s -check-prefix=cublasCdotu
// cublasCdotu: CUDA API:
// cublasCdotu-NEXT:   cublasCdotu(handle /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
// cublasCdotu-NEXT:               incx /*int*/, y /*const cuComplex **/, incy /*int*/,
// cublasCdotu-NEXT:               res /*cuComplex **/);
// cublasCdotu-NEXT: Is migrated to:
// cublasCdotu-NEXT:   sycl::float2* res_temp_ptr_ct{{[0-9]+}} = res;
// cublasCdotu-NEXT:   if(sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasCdotu-NEXT:     res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(1, dpct::get_default_queue());
// cublasCdotu-NEXT:   }
// cublasCdotu-NEXT:   oneapi::mkl::blas::column_major::dotu(handle->get_queue(), n, (std::complex<float>*)x, incx, (std::complex<float>*)y, incy, (std::complex<float>*)res_temp_ptr_ct{{[0-9]+}});
// cublasCdotu-NEXT:   if(sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(res, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
// cublasCdotu-NEXT:     handle->get_queue().wait();
// cublasCdotu-NEXT:     *res = *res_temp_ptr_ct{{[0-9]+}};
// cublasCdotu-NEXT:     sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
// cublasCdotu-NEXT:   }

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cublasDspr | FileCheck %s -check-prefix=cublasDspr
// cublasDspr: CUDA API:
// cublasDspr-NEXT:   cublasDspr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
// cublasDspr-NEXT:              n /*int*/, alpha /*const double **/, x /*const double **/,
// cublasDspr-NEXT:              incx /*int*/, a /*double **/);
// cublasDspr-NEXT: Is migrated to:
// cublasDspr-NEXT:   oneapi::mkl::blas::column_major::spr(handle->get_queue(), upper_lower, n, dpct::get_value(alpha, handle->get_queue()), x, incx, a);
