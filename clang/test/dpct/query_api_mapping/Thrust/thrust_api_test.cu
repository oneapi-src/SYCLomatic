
#include <hip/hip_runtime.h>
// UNSUPPORTED: system-windows
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_union_by_key --extra-arg="-std=c++14" | FileCheck %s -check-prefix=set_union_by_key
// set_union_by_key:  /*1*/ dpct::set_union(oneapi::dpl::execution::seq, A_keys, A_keys + 3, B_keys, B_keys + 4, A_vals, B_vals, keys_result, vals_result);
// set_union_by_key-NEXT:  /*2*/ dpct::set_union(oneapi::dpl::execution::seq, A_keys, A_keys + 3, B_keys, B_keys + 4, A_vals, B_vals, keys_result, vals_result);
// set_union_by_key-NEXT:  /*3*/ dpct::set_union(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result, Compare());
// set_union_by_key-NEXT:  /*4*/ dpct::set_union(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result, Compare());
// set_union_by_key-NEXT:  /*5*/ dpct::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin());
// set_union_by_key-NEXT:  /*6*/ dpct::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin());
// set_union_by_key-NEXT:  /*7*/ dpct::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin(), Compare());
// set_union_by_key-NEXT:  /*8*/ dpct::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin(), Compare());
// set_union_by_key-NEXT:  /*9*/ dpct::set_union(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin());
// set_union_by_key-NEXT:  /*10*/ dpct::set_union(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin());
// set_union_by_key-NEXT:  /*11*/ dpct::set_union(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin(), Compare());
// set_union_by_key-NEXT:  /*12*/ dpct::set_union(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin(), Compare());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_union --extra-arg="-std=c++14" | FileCheck %s -check-prefix=set_union
// set_union:  /*1*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, A1, A1 + 7, A2, A2 + 5, result);
// set_union-NEXT:  /*2*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, A1, A1 + 7, A2, A2 + 5, result);
// set_union-NEXT:  /*3*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, A1, A1 + 7, A2, A2 + 5, result, std::greater<int>());
// set_union-NEXT:  /*4*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, A1, A1 + 7, A2, A2 + 5, result, std::greater<int>());
// set_union-NEXT:  /*5*/ oneapi::dpl::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_union-NEXT:  /*6*/ oneapi::dpl::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_union-NEXT:  /*7*/ oneapi::dpl::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), std::greater<int>());
// set_union-NEXT:  /*8*/ oneapi::dpl::set_union(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), std::greater<int>());
// set_union-NEXT:  /*9*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_union-NEXT:  /*10*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_union-NEXT:  /*11*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), std::greater<int>());
// set_union-NEXT:  /*12*/ oneapi::dpl::set_union(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_intersection --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_intersection
// set_intersection:  /*1*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, A1, A1 + 6, A2, A2 + 7, result);
// set_intersection-NEXT:  /*2*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, A1, A1 + 6, A2, A2 + 7, result);
// set_intersection-NEXT:  /*3*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, A1, A1 + 6, A2, A2 + 7, result, std::greater<int>());
// set_intersection-NEXT:  /*4*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, A1, A1 + 6, A2, A2 + 7, result, std::greater<int>());
// set_intersection-NEXT:  /*5*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_intersection-NEXT:  /*6*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_intersection-NEXT:  /*7*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), std::greater<int>());
// set_intersection-NEXT:  /*8*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), std::greater<int>());
// set_intersection-NEXT:  /*9*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_intersection-NEXT:  /*10*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_intersection-NEXT:  /*11*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), std::greater<int>());
// set_intersection-NEXT:  /*12*/ oneapi::dpl::set_intersection(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::is_sorted_until --extra-arg="-std=c++14"| FileCheck %s -check-prefix=is_sorted_until
// is_sorted_until:  /*1*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, A, A + 8);
// is_sorted_until-NEXT:  /*2*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, A, A + 8);
// is_sorted_until-NEXT:  /*3*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, A, A + 8, comp);
// is_sorted_until-NEXT:  /*4*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, A, A + 8, comp);
// is_sorted_until-NEXT:  /*5*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, h_A.begin(), h_A.end());
// is_sorted_until-NEXT:  /*6*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, h_A.begin(), h_A.end());
// is_sorted_until-NEXT:  /*7*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, h_A.begin(), h_A.end(), comp);
// is_sorted_until-NEXT:  /*8*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::seq, h_A.begin(), h_A.end(), comp);
// is_sorted_until-NEXT:  /*9*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end());
// is_sorted_until-NEXT:  /*10*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end());
// is_sorted_until-NEXT:  /*11*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end(), comp);
// is_sorted_until-NEXT:  /*12*/ oneapi::dpl::is_sorted_until(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end(), comp);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::is_partitioned --extra-arg="-std=c++14"| FileCheck %s -check-prefix=is_partitioned
// is_partitioned:  /*1*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::seq, A, A + 10, is_even());
// is_partitioned-NEXT:  /*2*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::seq, A, A + 10, is_even());
// is_partitioned-NEXT:  /*3*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::seq, h_A.begin(), h_A.end(), is_even());
// is_partitioned-NEXT:  /*4*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::seq, h_A.begin(), h_A.end(), is_even());
// is_partitioned-NEXT:  /*5*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end(), is_even());
// is_partitioned-NEXT:  /*6*/ oneapi::dpl::is_partitioned(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.end(), is_even());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::none_of --extra-arg="-std=c++14"| FileCheck %s -check-prefix=none_of
// none_of:   /*1*/ oneapi::dpl::none_of(oneapi::dpl::execution::seq, A, A + 2, oneapi::dpl::identity());
// none_of-NEXT:   /*2*/ oneapi::dpl::none_of(oneapi::dpl::execution::seq, A, A + 2, oneapi::dpl::identity());
// none_of-NEXT:   /*3*/ oneapi::dpl::none_of(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + 2, oneapi::dpl::identity());
// none_of-NEXT:   /*4*/ oneapi::dpl::none_of(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + 2, oneapi::dpl::identity());
// none_of-NEXT:   /*5*/ oneapi::dpl::none_of(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + 2, oneapi::dpl::identity());
// none_of-NEXT:   /*6*/ oneapi::dpl::none_of(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + 2, oneapi::dpl::identity());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::all_of --extra-arg="-std=c++14"| FileCheck %s -check-prefix=all_of
// all_of: /*1*/  oneapi::dpl::all_of(oneapi::dpl::execution::seq, A, A + 2, oneapi::dpl::identity());
// all_of-NEXT: /*2*/  oneapi::dpl::all_of(oneapi::dpl::execution::seq, A, A + 2, oneapi::dpl::identity());
// all_of-NEXT: /*3*/  oneapi::dpl::all_of(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + 2, oneapi::dpl::identity());
// all_of-NEXT: /*4*/  oneapi::dpl::all_of(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + 2, oneapi::dpl::identity());
// all_of-NEXT: /*5*/  oneapi::dpl::all_of(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + 2, oneapi::dpl::identity());
// all_of-NEXT: /*6*/  oneapi::dpl::all_of(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + 2, oneapi::dpl::identity());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::uninitialized_fill_n --extra-arg="-std=c++14"| FileCheck %s -check-prefix=uninitialized_fill_n
// uninitialized_fill_n:  /*1*/ oneapi::dpl::uninitialized_fill_n(oneapi::dpl::execution::make_device_policy(q_ct1), d_array, N, int_val);
// uninitialized_fill_n-NEXT:  /*2*/ oneapi::dpl::uninitialized_fill_n(oneapi::dpl::execution::make_device_policy(q_ct1), d_array, N, int_val);
// uninitialized_fill_n-NEXT:  /*3*/ oneapi::dpl::uninitialized_fill_n(oneapi::dpl::execution::seq, data, N, val);
// uninitialized_fill_n-NEXT:  /*4*/ oneapi::dpl::uninitialized_fill_n(oneapi::dpl::execution::seq, data, N, val);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::swap_ranges --extra-arg="-std=c++14"| FileCheck %s -check-prefix=swap_ranges
// swap_ranges:  /*1*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::make_device_policy(q_ct1), d_v1.begin(), d_v1.end(), d_v2.begin());
// swap_ranges-NEXT:  /*2*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::make_device_policy(q_ct1), d_v1.begin(), d_v1.end(), d_v2.begin());
// swap_ranges-NEXT:  /*3*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::seq, h_v1.begin(), h_v1.end(), h_v2.begin());
// swap_ranges-NEXT:  /*4*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::seq, h_v1.begin(), h_v1.end(), h_v2.begin());
// swap_ranges-NEXT:  /*5*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::seq, v1, v1 + 2, v2);
// swap_ranges-NEXT:  /*6*/ oneapi::dpl::swap_ranges(oneapi::dpl::execution::seq, v1, v1 + 2, v2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_symmetric_difference_by_key --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_symmetric_difference_by_key
// set_symmetric_difference_by_key:  /*1*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result);
// set_symmetric_difference_by_key-NEXT:  /*2*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result);
// set_symmetric_difference_by_key-NEXT:  /*3*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result, Compare());
// set_symmetric_difference_by_key-NEXT:  /*4*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, A_keys, A_keys + 7, B_keys, B_keys + 5, A_vals, B_vals, keys_result, vals_result, Compare());
// set_symmetric_difference_by_key-NEXT:  /*5*/ dpct::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin());
// set_symmetric_difference_by_key-NEXT:  /*6*/ dpct::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin());
// set_symmetric_difference_by_key-NEXT:  /*7*/ dpct::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin(), Compare());
// set_symmetric_difference_by_key-NEXT:  /*8*/ dpct::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A_keys.begin(), d_A_keys.end(), d_B_keys.begin(), d_B_keys.end(), d_A_vals.begin(), d_B_vals.begin(), d_keys_result.begin(), d_vals_result.begin(), Compare());
// set_symmetric_difference_by_key-NEXT:  /*9*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin());
// set_symmetric_difference_by_key-NEXT:  /*10*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin());
// set_symmetric_difference_by_key-NEXT:  /*11*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin(), Compare());
// set_symmetric_difference_by_key-NEXT:  /*12*/ dpct::set_symmetric_difference(oneapi::dpl::execution::seq, h_A_keys.begin(), h_A_keys.end(), h_B_keys.begin(), h_B_keys.end(), h_A_vals.begin(), h_B_vals.begin(), h_keys_result.begin(), h_vals_result.begin(), Compare());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_symmetric_difference --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_symmetric_difference
// set_symmetric_difference:  /*1*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, A1, A1 + 4, A2, A2 + 2, result);
// set_symmetric_difference-NEXT:  /*2*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, A1, A1 + 4, A2, A2 + 2, result);
// set_symmetric_difference-NEXT:  /*3*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, A1, A1 + 5, A2, A2 + 5, result, Compare());
// set_symmetric_difference-NEXT:  /*4*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, A1, A1 + 5, A2, A2 + 5, result, Compare());
// set_symmetric_difference-NEXT:  /*5*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_symmetric_difference-NEXT:  /*6*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin());
// set_symmetric_difference-NEXT:  /*7*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), oneapi::dpl::less<int>());
// set_symmetric_difference-NEXT:  /*8*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin(), d_A2.end(), d_result.begin(), oneapi::dpl::less<int>());
// set_symmetric_difference-NEXT:  /*9*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_symmetric_difference-NEXT:  /*10*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin());
// set_symmetric_difference-NEXT:  /*11*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), oneapi::dpl::less<int>());
// set_symmetric_difference-NEXT:  /*12*/ oneapi::dpl::set_symmetric_difference(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin(), h_A2.end(), h_result.begin(), oneapi::dpl::less<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::equal --extra-arg="-std=c++14"| FileCheck %s -check-prefix=equal
// equal:  /*1*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, A1, A1 + N, A2);
// equal-NEXT:  /*2*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, A1, A1 + N, A2);
// equal-NEXT:  /*3*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, x, x + N, y, compare_modulo_two());
// equal-NEXT:  /*4*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, x, x + N, y, compare_modulo_two());
// equal-NEXT:  /*5*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin());
// equal-NEXT:  /*6*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, h_A1.begin(), h_A1.end(), h_A2.begin());
// equal-NEXT:  /*7*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, h_x.begin(), h_x.end(), h_y.begin(), compare_modulo_two());
// equal-NEXT:  /*8*/ oneapi::dpl::equal(oneapi::dpl::execution::seq, h_x.begin(), h_x.end(), h_y.begin(), compare_modulo_two());
// equal-NEXT:  /*9*/ oneapi::dpl::equal(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin());
// equal-NEXT:  /*10*/ oneapi::dpl::equal(oneapi::dpl::execution::make_device_policy(q_ct1), d_A1.begin(), d_A1.end(), d_A2.begin());
// equal-NEXT:  /*11*/ oneapi::dpl::equal(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), d_y.begin(), compare_modulo_two());
// equal-NEXT:  /*12*/ oneapi::dpl::equal(oneapi::dpl::execution::make_device_policy(q_ct1), d_x.begin(), d_x.end(), d_y.begin(), compare_modulo_two());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::uninitialized_copy_n --extra-arg="-std=c++14"| FileCheck %s -check-prefix=uninitialized_copy_n
// uninitialized_copy_n:  /*1*/ oneapi::dpl::uninitialized_copy_n(oneapi::dpl::execution::make_device_policy(q_ct1), d_input.begin(), N, d_array);
// uninitialized_copy_n-NEXT:  /*2*/ oneapi::dpl::uninitialized_copy_n(oneapi::dpl::execution::seq, h_data, N, h_array);
// uninitialized_copy_n-NEXT:  /*3*/ oneapi::dpl::uninitialized_copy_n(oneapi::dpl::execution::make_device_policy(q_ct1), d_input.begin(), N, d_array);
// uninitialized_copy_n-NEXT:  /*4*/ oneapi::dpl::uninitialized_copy_n(oneapi::dpl::execution::seq, h_data, N, h_array);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::uninitialized_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=uninitialized_copy
// uninitialized_copy:  /*1*/ oneapi::dpl::uninitialized_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_input.begin(), d_input.end(), d_array);
// uninitialized_copy-NEXT:  /*2*/ oneapi::dpl::uninitialized_copy(oneapi::dpl::execution::seq, data, data + N, array);
// uninitialized_copy-NEXT:  /*3*/ oneapi::dpl::uninitialized_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_input.begin(), d_input.end(), d_array);
// uninitialized_copy-NEXT:  /*4*/ oneapi::dpl::uninitialized_copy(oneapi::dpl::execution::seq, data, data + N, h_array);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::transform_inclusive_scan --extra-arg="-std=c++14"| FileCheck %s -check-prefix=transform_inclusive_scan
// transform_inclusive_scan:  /*1*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::seq, data, data + N, data, binary_op, unary_op);
// transform_inclusive_scan-NEXT:  /*2*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::seq, h_vec_data.begin(), h_vec_data.end(), h_vec_data.begin(), binary_op, unary_op);
// transform_inclusive_scan-NEXT:  /*3*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec_data.begin(), d_vec_data.end(), d_vec_data.begin(), binary_op, unary_op);
// transform_inclusive_scan-NEXT:  /*4*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::seq, data, data + N, data, binary_op, unary_op);
// transform_inclusive_scan-NEXT:  /*5*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::seq, h_vec_data.begin(), h_vec_data.end(), h_vec_data.begin(), binary_op, unary_op);
// transform_inclusive_scan-NEXT:  /*6*/ oneapi::dpl::transform_inclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), d_vec_data.begin(), d_vec_data.end(), d_vec_data.begin(), binary_op, unary_op);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::equal_range --extra-arg="-std=c++14"| FileCheck %s -check-prefix=equal_range
// equal_range:  /*1*/ dpct::equal_range(oneapi::dpl::execution::make_device_policy(q_ct1), device_vec.begin(), device_vec.end(), 0);
// equal_range-NEXT: /*2*/  dpct::equal_range(oneapi::dpl::execution::make_device_policy(q_ct1), device_vec.begin(), device_vec.end(), 0);
// equal_range-NEXT: /*3*/  dpct::equal_range(oneapi::dpl::execution::make_device_policy(q_ct1), device_vec.begin(), device_vec.end(), 0, oneapi::dpl::less<int>());
// equal_range-NEXT: /*4*/  dpct::equal_range(oneapi::dpl::execution::make_device_policy(q_ct1), device_vec.begin(), device_vec.end(), 0, oneapi::dpl::less<int>());
// equal_range-NEXT: /*5*/  dpct::equal_range(oneapi::dpl::execution::seq, host_vec.begin(), host_vec.end(), 0);
// equal_range-NEXT: /*6*/  dpct::equal_range(oneapi::dpl::execution::seq, host_vec.begin(), host_vec.end(), 0);
// equal_range-NEXT: /*7*/  dpct::equal_range(oneapi::dpl::execution::seq, host_vec.begin(), host_vec.end(), 0, oneapi::dpl::less<int>());
// equal_range-NEXT: /*8*/  dpct::equal_range(oneapi::dpl::execution::seq, host_vec.begin(), host_vec.end(), 0, oneapi::dpl::less<int>());
// equal_range-NEXT: /*9*/  dpct::equal_range(oneapi::dpl::execution::seq, data, data + N, 0);
// equal_range-NEXT: /*10*/  dpct::equal_range(oneapi::dpl::execution::seq, data, data + N, 0);
// equal_range-NEXT: /*11*/  dpct::equal_range(oneapi::dpl::execution::seq, data, data + N, 0, oneapi::dpl::less<int>());
// equal_range-NEXT: /*12*/  dpct::equal_range(oneapi::dpl::execution::seq, data, data + N, 0, oneapi::dpl::less<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::reverse --extra-arg="-std=c++14"| FileCheck %s -check-prefix=reverse
// reverse:   /*1*/ oneapi::dpl::reverse(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end());
// reverse-NEXT:  /*2*/ oneapi::dpl::reverse(oneapi::dpl::execution::seq, host_data.begin(), host_data.end());
// reverse-NEXT:  /*3*/ oneapi::dpl::reverse(oneapi::dpl::execution::seq, data, data + N);
// reverse-NEXT:  /*4*/ oneapi::dpl::reverse(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end());
// reverse-NEXT:  /*5*/ oneapi::dpl::reverse(oneapi::dpl::execution::seq, host_data.begin(), host_data.end());
// reverse-NEXT:  /*6*/ oneapi::dpl::reverse(oneapi::dpl::execution::seq, data, data + N);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::replace_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=replace_copy
// replace_copy:  /*1*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_data.begin(), d_data.end(), d_result.begin(), 1, 99);
// replace_copy-NEXT:  /*2*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::seq, h_data.begin(), h_data.end(), h_result.begin(), 1, 99);
// replace_copy-NEXT:  /*3*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::seq, data, data + N, result, 1, 99);
// replace_copy-NEXT:  /*4*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_data.begin(), d_data.end(), d_result.begin(), 1, 99);
// replace_copy-NEXT:  /*5*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::seq, h_data.begin(), h_data.end(), h_result.begin(), 1, 99);
// replace_copy-NEXT:  /*6*/ oneapi::dpl::replace_copy(oneapi::dpl::execution::seq, data, data + N, result, 1, 99);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::mismatch --extra-arg="-std=c++14"| FileCheck %s -check-prefix=mismatch
// mismatch:  /*1*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, VA.begin(), VA.end(), VB.begin());
// mismatch-NEXT:  /*2*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, VA.begin(), VA.end(), VB.begin());
// mismatch-NEXT:  /*3*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, VA.begin(), VA.end(), VB.begin(), oneapi::dpl::equal_to<int>());
// mismatch-NEXT:  /*4*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, VA.begin(), VA.end(), VB.begin(), oneapi::dpl::equal_to<int>());
// mismatch-NEXT:  /*5*/ oneapi::dpl::mismatch(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin());
// mismatch-NEXT:  /*6*/ oneapi::dpl::mismatch(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin());
// mismatch-NEXT:  /*7*/ oneapi::dpl::mismatch(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), oneapi::dpl::equal_to<int>());
// mismatch-NEXT:  /*8*/ oneapi::dpl::mismatch(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), oneapi::dpl::equal_to<int>());
// mismatch-NEXT:  /*9*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, A, A + N, B);
// mismatch-NEXT:  /*10*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, A, A + N, B);
// mismatch-NEXT:  /*11*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, A, A + N, B, oneapi::dpl::equal_to<int>());
// mismatch-NEXT:  /*12*/ oneapi::dpl::mismatch(oneapi::dpl::execution::seq, A, A + N, B, oneapi::dpl::equal_to<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::find_if_not --extra-arg="-std=c++14"| FileCheck %s -check-prefix=find_if_not
// find_if_not:  /*1*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::seq, data, data + 3, greater_than_four());
// find_if_not-NEXT:  /*2*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), greater_than_four());
// find_if_not-NEXT:  /*3*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), greater_than_four());
// find_if_not-NEXT:  /*4*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::seq, data, data + 3, greater_than_four());
// find_if_not-NEXT:  /*5*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), greater_than_four());
// find_if_not-NEXT:  /*6*/ oneapi::dpl::find_if_not(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), greater_than_four());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::find_if --extra-arg="-std=c++14"| FileCheck %s -check-prefix=find_if
// find_if:  /*1*/ oneapi::dpl::find_if(oneapi::dpl::execution::seq, data, data + 3, greater_than_four());
// find_if-NEXT:  /*2*/ oneapi::dpl::find_if(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), greater_than_four());
// find_if-NEXT:  /*3*/ oneapi::dpl::find_if(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), greater_than_four());
// find_if-NEXT:  /*4*/ oneapi::dpl::find_if(oneapi::dpl::execution::seq, data, data + 3, greater_than_four());
// find_if-NEXT:  /*5*/ oneapi::dpl::find_if(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), greater_than_four());
// find_if-NEXT:  /*6*/ oneapi::dpl::find_if(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), greater_than_four());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::remove --extra-arg="-std=c++14"| FileCheck %s -check-prefix=remove
// remove:  /*1*/ oneapi::dpl::remove(oneapi::dpl::execution::seq, data, data + N, 1);
// remove-NEXT:  /*2*/ oneapi::dpl::remove(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, 1);
// remove-NEXT:  /*3*/ oneapi::dpl::remove(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, 1);
// remove-NEXT:  /*4*/ oneapi::dpl::remove(oneapi::dpl::execution::seq, data, data + N, 1);
// remove-NEXT:  /*5*/ oneapi::dpl::remove(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, 1);
// remove-NEXT:  /*6*/ oneapi::dpl::remove(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, 1);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::stable_partition_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=stable_partition_copy
// stable_partition_copy:  /*1*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, data, data + N, evens, odds, is_even());
// stable_partition_copy-NEXT:  /*2*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, host_a.begin(), host_a.begin() + N, host_evens.begin(), host_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*3*/ dpct::stable_partition_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_a.begin(), device_a.begin() + N, device_evens.begin(), device_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*4*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, data, data + N, evens, odds, is_even());
// stable_partition_copy-NEXT:  /*5*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, host_a.begin(), host_a.begin() + N, host_evens.begin(), host_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*6*/ dpct::stable_partition_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_a.begin(), device_a.begin() + N, device_evens.begin(), device_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*7*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, data, data + N, S, evens, odds, is_even());
// stable_partition_copy-NEXT:  /*8*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, host_a.begin(), host_a.begin() + N, host_S.begin(), host_evens.begin(), host_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*9*/ dpct::stable_partition_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_a.begin(), device_a.begin() + N, device_S.begin(), device_evens.begin(), device_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*10*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, data, data + N, S, evens, odds, is_even());
// stable_partition_copy-NEXT:  /*11*/ dpct::stable_partition_copy(oneapi::dpl::execution::seq, host_a.begin(), host_a.begin() + N, host_S.begin(), host_evens.begin(), host_odds.begin(), is_even());
// stable_partition_copy-NEXT:  /*12*/ dpct::stable_partition_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_a.begin(), device_a.begin() + N, device_S.begin(), device_evens.begin(), device_odds.begin(), is_even());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::stable_partition --extra-arg="-std=c++14"| FileCheck %s -check-prefix=stable_partition
// stable_partition:  /*1*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::seq, data, data + N, is_even());
// stable_partition-NEXT:  /*2*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, is_even());
// stable_partition-NEXT:  /*3*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, is_even());
// stable_partition-NEXT:  /*4*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::seq, data, data + N, is_even());
// stable_partition-NEXT:  /*5*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, is_even());
// stable_partition-NEXT:  /*6*/ oneapi::dpl::stable_partition(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, is_even());
// stable_partition-NEXT:  /*7*/ dpct::stable_partition(oneapi::dpl::execution::seq, data, data + N, S, is_even());
// stable_partition-NEXT:  /*8*/ dpct::stable_partition(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, host_S.begin(), is_even());
// stable_partition-NEXT:  /*9*/ dpct::stable_partition(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, device_s.begin(), is_even());
// stable_partition-NEXT:  /*10*/ dpct::stable_partition(oneapi::dpl::execution::seq, data, data + N, S, is_even());
// stable_partition-NEXT:  /*11*/ dpct::stable_partition(oneapi::dpl::execution::seq, host_data.begin(), host_data.begin() + N, host_S.begin(), is_even());
// stable_partition-NEXT:  /*12*/ dpct::stable_partition(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.begin() + N, device_s.begin(), is_even());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::scatter_if --extra-arg="-std=c++14"| FileCheck %s -check-prefix=scatter_if
// scatter_if:  /*1*/ dpct::scatter_if(oneapi::dpl::execution::seq, V, V + 8, M, S, D);
// scatter_if-NEXT:  /*2*/ dpct::scatter_if(oneapi::dpl::execution::seq, V, V + 8, M, S, D);
// scatter_if-NEXT:  /*3*/ dpct::scatter_if(oneapi::dpl::execution::seq, V, V + 8, M, S, D, pred);
// scatter_if-NEXT:  /*4*/ dpct::scatter_if(oneapi::dpl::execution::seq, V, V + 8, M, S, D, pred);
// scatter_if-NEXT:  /*5*/  dpct::scatter_if(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_M.begin(), d_S.begin(), d_D.begin());
// scatter_if-NEXT:  /*6*/  dpct::scatter_if(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_M.begin(), d_S.begin(), d_D.begin());
// scatter_if-NEXT:  /*7*/  dpct::scatter_if(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_M.begin(), d_S.begin(), d_D.begin(), pred);
// scatter_if-NEXT:  /*8*/  dpct::scatter_if(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_M.begin(), d_S.begin(), d_D.begin(), pred);
// scatter_if-NEXT:  /*9*/  dpct::scatter_if(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_M.begin(), h_S.begin(), h_D.begin());
// scatter_if-NEXT:  /*10*/  dpct::scatter_if(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_M.begin(), h_S.begin(), h_D.begin());
// scatter_if-NEXT:  /*11*/  dpct::scatter_if(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_M.begin(), h_S.begin(), h_D.begin(), pred);
// scatter_if-NEXT:  /*12*/  dpct::scatter_if(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_M.begin(), h_S.begin(), h_D.begin(), pred);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::reverse_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=reverse_copy
// reverse_copy:  /*1*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), device_result.begin());
// reverse_copy-NEXT:  /*2*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), host_result.begin());
// reverse_copy-NEXT:  /*3*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::seq, data, data + N, result);
// reverse_copy-NEXT:  /*4*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::make_device_policy(q_ct1), device_data.begin(), device_data.end(), device_result.begin());
// reverse_copy-NEXT:  /*5*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::seq, host_data.begin(), host_data.end(), host_result.begin());
// reverse_copy-NEXT:  /*6*/  oneapi::dpl::reverse_copy(oneapi::dpl::execution::seq, data, data + N, result);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::unique_count --extra-arg="-std=c++14"| FileCheck %s -check-prefix=unique_count
// unique_count:  /*1*/  count = dpct::unique_count(oneapi::dpl::execution::seq, A, A + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*2*/  count = dpct::unique_count(oneapi::dpl::execution::seq, A, A + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*3*/  count = dpct::unique_count(oneapi::dpl::execution::seq, A, A + N);
// unique_count-NEXT:  /*4*/  count = dpct::unique_count(oneapi::dpl::execution::seq, A, A + N);
// unique_count-NEXT:  /*5*/  count = dpct::unique_count(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*6*/  count = dpct::unique_count(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*7*/  count = dpct::unique_count(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*8*/  count = dpct::unique_count(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + N, oneapi::dpl::equal_to<int>());
// unique_count-NEXT:  /*9*/  count = dpct::unique_count(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + N);
// unique_count-NEXT:  /*10*/  count = dpct::unique_count(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + N);
// unique_count-NEXT:  /*11*/  count = dpct::unique_count(oneapi::dpl::execution::seq, h_A.begin(), h_A.begin() + N);
// unique_count-NEXT:  /*12*/  count = dpct::unique_count(oneapi::dpl::execution::make_device_policy(q_ct1), d_A.begin(), d_A.begin() + N);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::partition_point --extra-arg="-std=c++14"| FileCheck %s -check-prefix=partition_point
// partition_point:  /*1*/  dpct::partition_point(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), up);
// partition_point-NEXT:  /*2*/  dpct::partition_point(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), up);
// partition_point-NEXT:  /*3*/  dpct::partition_point(oneapi::dpl::execution::seq, data, data + 10, up);
// partition_point-NEXT:  /*4*/  dpct::partition_point(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), up);
// partition_point-NEXT:  /*5*/  dpct::partition_point(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), up);
// partition_point-NEXT:  /*6*/  dpct::partition_point(oneapi::dpl::execution::seq, data, data + 10, up);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::minmax_element --extra-arg="-std=c++14"| FileCheck %s -check-prefix=minmax_element
// minmax_element:  /*1*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, h_values.begin(), h_values.end());
// minmax_element-NEXT:  /*2*/ oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, h_values.begin(), h_values.end());
// minmax_element-NEXT:  /*3*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, h_values.begin(), h_values.begin() + 4, compare_key_value());
// minmax_element-NEXT:  /*4*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, h_values.begin(), h_values.begin() + 4, compare_key_value());
// minmax_element-NEXT:  /*5*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end());
// minmax_element-NEXT:  /*6*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end());
// minmax_element-NEXT:  /*7*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end(), compare_key_value());
// minmax_element-NEXT:  /*8*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::make_device_policy(q_ct1), d_values.begin(), d_values.end(), compare_key_value());
// minmax_element-NEXT:  /*9*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, data, data+N);
// minmax_element-NEXT:  /*10*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, data, data+N);
// minmax_element-NEXT:  /*11*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, data, data+N, compare_key_value());
// minmax_element-NEXT:  /*12*/  oneapi::dpl::minmax_element(oneapi::dpl::execution::seq, data, data+N, compare_key_value());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::unique_by_key --extra-arg="-std=c++14"| FileCheck %s -check-prefix=unique_by_key
// unique_by_key:  /*1*/ dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin());
// unique_by_key-NEXT:  /*2*/ dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin());
// unique_by_key-NEXT:   /*3*/ dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), binary_pred);
// unique_by_key-NEXT:   /*4*/ dpct::unique(oneapi::dpl::execution::seq, h_keys.begin(), h_keys.end(), h_values.begin(), binary_pred);
// unique_by_key-NEXT:   /*5*/ dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin());
// unique_by_key-NEXT:   /*6*/ dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin());
// unique_by_key-NEXT:   /*7*/ dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
// unique_by_key-NEXT:   /*8*/ dpct::unique(oneapi::dpl::execution::make_device_policy(q_ct1), d_keys.begin(), d_keys.end(), d_values.begin(), binary_pred);
// unique_by_key-NEXT:   /*9*/ dpct::unique(oneapi::dpl::execution::seq, A, A + N, B);
// unique_by_key-NEXT:   /*10*/ dpct::unique(oneapi::dpl::execution::seq, A, A + N, B);
// unique_by_key-NEXT:   /*11*/ dpct::unique(oneapi::dpl::execution::seq, A, A + N, B, binary_pred);
// unique_by_key-NEXT:   /*12*/ dpct::unique(oneapi::dpl::execution::seq, A, A + N, B, binary_pred);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::is_sorted --extra-arg="-std=c++14"| FileCheck %s -check-prefix=is_sorted
// is_sorted:  /*1*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, h_v.begin(), h_v.end());
// is_sorted-NEXT:  /*2*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, h_v.begin(), h_v.end());
// is_sorted-NEXT:  /*3*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), comp);
// is_sorted-NEXT:  /*4*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), comp);
// is_sorted-NEXT:  /*5*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), comp);
// is_sorted-NEXT:  /*6*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end());
// is_sorted-NEXT:  /*7*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), comp);
// is_sorted-NEXT:  /*8*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), comp);
// is_sorted-NEXT:  /*9*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, datas, datas + N);
// is_sorted-NEXT:  /*10*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, datas, datas + N);
// is_sorted-NEXT:  /*11*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, datas, datas + N, comp);
// is_sorted-NEXT:  /*12*/ oneapi::dpl::is_sorted(oneapi::dpl::execution::seq, datas, datas + N, comp);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::partition --extra-arg="-std=c++14"| FileCheck %s -check-prefix=partition
// partition:  /*1*/ oneapi::dpl::partition(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), is_even());
// partition-NEXT:  /*2*/ oneapi::dpl::partition(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), is_even());
// partition-NEXT:  /*3*/ dpct::partition(oneapi::dpl::execution::seq, h_vdata.begin(), h_vdata.end(), h_vstencil.begin(), is_even());
// partition-NEXT:  /*4*/ dpct::partition(oneapi::dpl::execution::seq, h_vdata.begin(), h_vdata.end(), h_vstencil.begin(), is_even());
// partition-NEXT:  /*5*/ oneapi::dpl::partition(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), is_even());
// partition-NEXT:  /*6*/ oneapi::dpl::partition(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), is_even());
// partition-NEXT:  /*7*/ dpct::partition(oneapi::dpl::execution::make_device_policy(q_ct1), d_vdata.begin(), d_vdata.end(), d_vstencil.begin(), is_even());
// partition-NEXT:  /*8*/ dpct::partition(oneapi::dpl::execution::make_device_policy(q_ct1), d_vdata.begin(), d_vdata.end(), d_vstencil.begin(), is_even());
// partition-NEXT:  /*9*/ oneapi::dpl::partition(oneapi::dpl::execution::seq, datas, datas + N, is_even());
// partition-NEXT:  /*10*/ oneapi::dpl::partition(oneapi::dpl::execution::seq, datas, datas + N, is_even());
// partition-NEXT:  /*11*/ dpct::partition(oneapi::dpl::execution::seq, datas, datas + N, stencil, is_even());
// partition-NEXT:  /*12*/ dpct::partition(oneapi::dpl::execution::seq, datas, datas + N, stencil, is_even());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::unique_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=unique_copy
// unique_copy:  /*1*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin());
// unique_copy-NEXT:  /*2*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin());
// unique_copy-NEXT:  /*3*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin(), oneapi::dpl::equal_to<int>());
// unique_copy-NEXT:  /*4*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin(), oneapi::dpl::equal_to<int>());
// unique_copy-NEXT:  /*5*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin());
// unique_copy-NEXT:  /*6*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin());
// unique_copy-NEXT:  /*7*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin(), oneapi::dpl::equal_to<int>());
// unique_copy-NEXT:  /*8*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin(), oneapi::dpl::equal_to<int>());
// unique_copy-NEXT:  /*9*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, A, A + N, B);
// unique_copy-NEXT:  /*10*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, A, A + N, B);
// unique_copy-NEXT:  /*11*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, A, A + N, B, oneapi::dpl::equal_to<int>());
// unique_copy-NEXT:  /*12*/ oneapi::dpl::unique_copy(oneapi::dpl::execution::seq, A, A + N, B, oneapi::dpl::equal_to<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::stable_sort --extra-arg="-std=c++14"| FileCheck %s -check-prefix=stable_sort
// stable_sort:  /*1*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, h_v.begin(), h_v.end());
// stable_sort-NEXT:  /*2*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, h_v.begin(), h_v.end());
// stable_sort-NEXT:  /*3*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), std::greater<int>());
// stable_sort-NEXT:  /*4*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, h_v.begin(), h_v.end(), std::greater<int>());
// stable_sort-NEXT:  /*5*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end());
// stable_sort-NEXT:  /*6*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end());
// stable_sort-NEXT:  /*7*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), std::greater<int>());
// stable_sort-NEXT:  /*8*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::make_device_policy(q_ct1), d_v.begin(), d_v.end(), std::greater<int>());
// stable_sort-NEXT:  /*9*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, datas, datas + N);
// stable_sort-NEXT:  /*10*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, datas, datas + N);
// stable_sort-NEXT:  /*11*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, datas, datas + N, std::greater<int>());
// stable_sort-NEXT:  /*12*/ oneapi::dpl::stable_sort(oneapi::dpl::execution::seq, datas, datas + N, std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_difference_by_key --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_difference_by_key
// set_difference_by_key:  /*1*/ dpct::set_difference(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VBvalue.begin(), h_VCkey.begin(), h_VCvalue.begin());
// set_difference_by_key-NEXT:  /*2*/ dpct::set_difference(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VBvalue.begin(), h_VCkey.begin(), h_VCvalue.begin());
// set_difference_by_key-NEXT:  /*3*/ dpct::set_difference(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VBvalue.begin(), h_VCkey.begin(), h_VCvalue.begin(), std::greater<int>());
// set_difference_by_key-NEXT:  /*4*/ dpct::set_difference(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VBvalue.begin(), h_VCkey.begin(), h_VCvalue.begin(), std::greater<int>());
// set_difference_by_key-NEXT:  /*5*/ dpct::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VBvalue.begin(), d_VCkey.begin(), d_VCvalue.begin());
// set_difference_by_key-NEXT:  /*6*/ dpct::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VBvalue.begin(), d_VCkey.begin(), d_VCvalue.begin());
// set_difference_by_key-NEXT:  /*7*/ dpct::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VBvalue.begin(), d_VCkey.begin(), d_VCvalue.begin(), std::greater<int>());
// set_difference_by_key-NEXT:  /*8*/ dpct::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VBvalue.begin(), d_VCkey.begin(), d_VCvalue.begin(), std::greater<int>());
// set_difference_by_key-NEXT:  /*9*/ dpct::set_difference(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Bvalue, Ckey, Cvalue);
// set_difference_by_key-NEXT:  /*10*/ dpct::set_difference(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Bvalue, Ckey, Cvalue);
// set_difference_by_key-NEXT:  /*11*/ dpct::set_difference(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Bvalue, Ckey, Cvalue, std::greater<int>());
// set_difference_by_key-NEXT:  /*12*/ dpct::set_difference(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Bvalue, Ckey, Cvalue, std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_difference --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_difference
// set_difference:  /*1*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, h_VA.begin(), h_VA.end(), h_VB.begin(), h_VB.end(), h_VC.begin());
// set_difference-NEXT:  /*2*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, h_VA.begin(), h_VA.end(), h_VB.begin(), h_VB.end(), h_VC.begin());
// set_difference-NEXT:  /*3*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, h_VA.begin(), h_VA.end(), h_VB.begin(), h_VB.end(), h_VC.begin(), std::greater<int>());
// set_difference-NEXT:  /*4*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, h_VA.begin(), h_VA.end(), h_VB.begin(), h_VB.end(), h_VC.begin(), std::greater<int>());
// set_difference-NEXT:  /*5*/ oneapi::dpl::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), d_VB.end(), d_VC.begin());
// set_difference-NEXT:  /*6*/ oneapi::dpl::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), d_VB.end(), d_VC.begin());
// set_difference-NEXT:  /*7*/ oneapi::dpl::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), d_VB.end(), d_VC.begin(), std::greater<int>());
// set_difference-NEXT:  /*8*/ oneapi::dpl::set_difference(oneapi::dpl::execution::make_device_policy(q_ct1), d_VA.begin(), d_VA.end(), d_VB.begin(), d_VB.end(), d_VC.begin(), std::greater<int>());
// set_difference-NEXT:  /*9*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, A, A + N, B, B + M, C);
// set_difference-NEXT:  /*10*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, A, A + N, B, B + M, C);
// set_difference-NEXT:  /*11*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, A, A + N, B, B + M, C, std::greater<int>());
// set_difference-NEXT:  /*12*/ oneapi::dpl::set_difference(oneapi::dpl::execution::seq, A, A + N, B, B + M, C, std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::for_each_n --extra-arg="-std=c++14"| FileCheck %s -check-prefix=for_each_n
// for_each_n:  /*1*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::seq, h_V.begin(), h_V.size(), add_functor());
// for_each_n-NEXT:  /*2*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::seq, h_V.begin(), h_V.size(), add_functor());
// for_each_n-NEXT:  /*3*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.size(), add_functor());
// for_each_n-NEXT:  /*4*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::seq, d_V.begin(), d_V.size(), add_functor());
// for_each_n-NEXT:  /*5*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::seq, A, N, add_functor());
// for_each_n-NEXT:  /*6*/ oneapi::dpl::for_each_n(oneapi::dpl::execution::seq, A, N, add_functor());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::tabulate --extra-arg="-std=c++14"| FileCheck %s -check-prefix=tabulate
// tabulate:  /*1*/ dpct::for_each_index(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), std::negate<int>());
// tabulate-NEXT:  /*2*/ dpct::for_each_index(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), std::negate<int>());
// tabulate-NEXT:  /*3*/ dpct::for_each_index(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), std::negate<int>());
// tabulate-NEXT:  /*4*/ dpct::for_each_index(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), std::negate<int>());
// tabulate-NEXT:  /*5*/ dpct::for_each_index(oneapi::dpl::execution::seq, A, A + N, std::negate<int>());
// tabulate-NEXT:  /*6*/ dpct::for_each_index(oneapi::dpl::execution::seq, A, A + N, std::negate<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::remove_copy --extra-arg="-std=c++14"| FileCheck %s -check-prefix=remove_copy
// remove_copy:  /*1*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin(), 0);
// remove_copy-NEXT:  /*2*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_result.begin(), 0);
// remove_copy-NEXT:  /*3*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin(), 0);
// remove_copy-NEXT:  /*4*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_result.begin(), 0);
// remove_copy-NEXT:  /*5*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::seq, V, V + N, result, 0);
// remove_copy-NEXT:  /*6*/ oneapi::dpl::remove_copy(oneapi::dpl::execution::seq, V, V + N, result, 0);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::transform_exclusive_scan --extra-arg="-std=c++14"| FileCheck %s -check-prefix=transform_exclusive_scan
// transform_exclusive_scan:  /*1*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_V.begin(), 4, binary_op, unary_op);
// transform_exclusive_scan-NEXT:  /*2*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::seq, h_V.begin(), h_V.end(), h_V.begin(), 4, binary_op, unary_op);
// transform_exclusive_scan-NEXT:  /*3*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_V.begin(), 4, binary_op, unary_op);
// transform_exclusive_scan-NEXT:  /*4*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::make_device_policy(q_ct1), d_V.begin(), d_V.end(), d_V.begin(), 4, binary_op, unary_op);
// transform_exclusive_scan-NEXT:  /*5*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::seq, A, A + N, A, 4, binary_op, unary_op);
// transform_exclusive_scan-NEXT:  /*6*/ oneapi::dpl::transform_exclusive_scan(oneapi::dpl::execution::seq, A, A + N, A, 4, binary_op, unary_op);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::set_intersection_by_key --extra-arg="-std=c++14"| FileCheck %s -check-prefix=set_intersection_by_key
// set_intersection_by_key:  /*1*/ dpct::set_intersection(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VCkey.begin(), h_VCvalue.begin());
// set_intersection_by_key-NEXT:  /*2*/ dpct::set_intersection(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VCkey.begin(), h_VCvalue.begin());
// set_intersection_by_key-NEXT:  /*3*/ dpct::set_intersection(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VCkey.begin(), h_VCvalue.begin(), std::greater<int>());
// set_intersection_by_key-NEXT:  /*4*/ dpct::set_intersection(oneapi::dpl::execution::seq, h_VAkey.begin(), h_VAkey.end(), h_VBkey.begin(), h_VBkey.end(), h_VAvalue.begin(), h_VCkey.begin(), h_VCvalue.begin(), std::greater<int>());
// set_intersection_by_key-NEXT:  /*5*/ dpct::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VCkey.begin(), d_VCvalue.begin());
// set_intersection_by_key-NEXT:  /*6*/ dpct::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VCkey.begin(), d_VCvalue.begin());
// set_intersection_by_key-NEXT:  /*7*/ dpct::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VCkey.begin(), d_VCvalue.begin(), std::greater<int>());
// set_intersection_by_key-NEXT:  /*8*/ dpct::set_intersection(oneapi::dpl::execution::make_device_policy(q_ct1), d_VAkey.begin(), d_VAkey.end(), d_VBkey.begin(), d_VBkey.end(), d_VAvalue.begin(), d_VCkey.begin(), d_VCvalue.begin(), std::greater<int>());
// set_intersection_by_key-NEXT:  /*9*/ dpct::set_intersection(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Ckey, Cvalue);
// set_intersection_by_key-NEXT:  /*10*/ dpct::set_intersection(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Ckey, Cvalue);
// set_intersection_by_key-NEXT:  /*11*/ dpct::set_intersection(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Ckey, Cvalue, std::greater<int>());
// set_intersection_by_key-NEXT:  /*12*/ dpct::set_intersection(oneapi::dpl::execution::seq, Akey, Akey + N, Bkey, Bkey + M, Avalue, Ckey, Cvalue, std::greater<int>());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=thrust::raw_reference_cast --extra-arg="-std=c++14"| FileCheck %s -check-prefix=raw_reference_cast
// raw_reference_cast:  /*1*/ int &ref1 = dpct::get_raw_reference(d_vec[0]);
// raw_reference_cast-NEXT:  /*2*/ int &ref2 = dpct::get_raw_reference(ref_const);

