
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cusparseSpMM_preprocess | FileCheck %s -check-prefix=cusparseSpMM_preprocess
// cusparseSpMM_preprocess: CUDA API:
// cusparseSpMM_preprocess-NEXT:   cusparseSpMM_preprocess(
// cusparseSpMM_preprocess-NEXT:       handle /*cusparseHandle_t*/, transa /*cusparseOperation_t*/,
// cusparseSpMM_preprocess-NEXT:       transb /*cusparseOperation_t*/, alpha /*const void **/,
// cusparseSpMM_preprocess-NEXT:       a /*cusparseConstSpMatDescr_t*/, b /*cusparseConstDnMatDescr_t*/,
// cusparseSpMM_preprocess-NEXT:       beta /*const void **/, c /*cusparseDnMatDescr_t*/,
// cusparseSpMM_preprocess-NEXT:       computetype /*cudaDataType*/, algo /*cusparseSpMMAlg_t*/,
// cusparseSpMM_preprocess-NEXT:       workspace /*void **/);
// cusparseSpMM_preprocess-NEXT: Is migrated to:
// cusparseSpMM_preprocess-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cusparseSpMM_bufferSize | FileCheck %s -check-prefix=cusparseSpMM_bufferSize
// cusparseSpMM_bufferSize: CUDA API:
// cusparseSpMM_bufferSize-NEXT:   cusparseSpMM_bufferSize(
// cusparseSpMM_bufferSize-NEXT:       handle /*cusparseHandle_t*/, transa /*cusparseOperation_t*/,
// cusparseSpMM_bufferSize-NEXT:       transb /*cusparseOperation_t*/, alpha /*const void **/,
// cusparseSpMM_bufferSize-NEXT:       a /*cusparseConstSpMatDescr_t*/, b /*cusparseConstDnMatDescr_t*/,
// cusparseSpMM_bufferSize-NEXT:       beta /*const void **/, c /*cusparseDnMatDescr_t*/,
// cusparseSpMM_bufferSize-NEXT:       computetype /*cudaDataType*/, algo /*cusparseSpMMAlg_t*/,
// cusparseSpMM_bufferSize-NEXT:       workspace_size /*size_t **/);
// cusparseSpMM_bufferSize-NEXT: Is migrated to:
// cusparseSpMM_bufferSize-NEXT:   *workspace_size = 0;
// cusparseSpMM_bufferSize-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cusparseSpMM | FileCheck %s -check-prefix=cusparseSpMM
// cusparseSpMM: CUDA API:
// cusparseSpMM-NEXT:   cusparseSpMM(handle /*cusparseHandle_t*/, transa /*cusparseOperation_t*/,
// cusparseSpMM-NEXT:                transb /*cusparseOperation_t*/, alpha /*const void **/,
// cusparseSpMM-NEXT:                a /*cusparseConstSpMatDescr_t*/, b /*cusparseConstDnMatDescr_t*/,
// cusparseSpMM-NEXT:                beta /*const void **/, c /*cusparseDnMatDescr_t*/,
// cusparseSpMM-NEXT:                computetype /*cudaDataType*/, algo /*cusparseSpMMAlg_t*/,
// cusparseSpMM-NEXT:                workspace /*void **/);
// cusparseSpMM-NEXT: Is migrated to:
// cusparseSpMM-NEXT:   dpct::sparse::spmm(*handle, transa, transb, alpha, a, b, beta, c, computetype);
// cusparseSpMM-EMPTY:

