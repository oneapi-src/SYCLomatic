
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=skipahead | FileCheck %s -check-prefix=SKIPAHEAD
// SKIPAHEAD: CUDA API:
// SKIPAHEAD-NEXT:   curandStateMRG32k3a_t *ps1;
// SKIPAHEAD-NEXT:   curandStatePhilox4_32_10_t *ps2;
// SKIPAHEAD-NEXT:   curandStateXORWOW_t *ps3;
// SKIPAHEAD-NEXT:   /* 1 */ skipahead(ull, ps1 /*curandStateMRG32k3a_t **/);
// SKIPAHEAD-NEXT:   /* 2 */ skipahead(ull, ps2 /*curandStatePhilox4_32_10_t **/);
// SKIPAHEAD-NEXT:   /* 3 */ skipahead(ull, ps3 /*curandStateXORWOW_t **/);
// SKIPAHEAD-NEXT:   /* 4 */ skipahead(u, ps1 /*curandStateMRG32k3a_t **/);
// SKIPAHEAD-NEXT:   /* 5 */ skipahead(u, ps2 /*curandStatePhilox4_32_10_t **/);
// SKIPAHEAD-NEXT:   /* 6 */ skipahead(u, ps3 /*curandStateXORWOW_t **/);
// SKIPAHEAD-NEXT: Is migrated to:
// SKIPAHEAD-NEXT:   dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mrg32k3a<1>> *ps1;
// SKIPAHEAD-NEXT:   dpct::rng::device::rng_generator<oneapi::mkl::rng::device::philox4x32x10<1>> *ps2;
// SKIPAHEAD-NEXT:   dpct::rng::device::rng_generator<oneapi::mkl::rng::device::mcg59<1>> *ps3;
// SKIPAHEAD-NEXT:   /* 1 */ oneapi::mkl::rng::device::skip_ahead(ps1->get_engine(), ull);
// SKIPAHEAD-NEXT:   /* 2 */ oneapi::mkl::rng::device::skip_ahead(ps2->get_engine(), ull);
// SKIPAHEAD-NEXT:   /* 3 */ oneapi::mkl::rng::device::skip_ahead(ps3->get_engine(), ull);
// SKIPAHEAD-NEXT:   /* 4 */ oneapi::mkl::rng::device::skip_ahead(ps1->get_engine(), u);
// SKIPAHEAD-NEXT:   /* 5 */ oneapi::mkl::rng::device::skip_ahead(ps2->get_engine(), u);
// SKIPAHEAD-NEXT:   /* 6 */ oneapi::mkl::rng::device::skip_ahead(ps3->get_engine(), u);
