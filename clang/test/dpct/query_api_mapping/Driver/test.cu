
#include <hip/hip_runtime.h>
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0

/// Initialization

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuInit | FileCheck %s -check-prefix=CUINIT
// CUINIT: CUDA API:
// CUINIT-NEXT:   cuInit(u /*unsigned int*/);
// CUINIT-NEXT: The API is Removed.

/// Version Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDriverGetVersion | FileCheck %s -check-prefix=CUDRIVERGETVERSION
// CUDRIVERGETVERSION: CUDA API:
// CUDRIVERGETVERSION-NEXT:   cuDriverGetVersion(pi /*int **/);
// CUDRIVERGETVERSION-NEXT: Is migrated to:
// CUDRIVERGETVERSION-NEXT:   *pi = std::stoi(dpct::get_current_device().get_info<sycl::info::device::version>());

/// Device Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGet | FileCheck %s -check-prefix=CUDEVICEGET
// CUDEVICEGET: CUDA API:
// CUDEVICEGET-NEXT:   cuDeviceGet(pd /*CUdevice **/, i /*int*/);
// CUDEVICEGET-NEXT: Is migrated to:
// CUDEVICEGET-NEXT:   *pd = i;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGetCount | FileCheck %s -check-prefix=CUDEVICEGETCOUNT
// CUDEVICEGETCOUNT: CUDA API:
// CUDEVICEGETCOUNT-NEXT:   cuDeviceGetCount(pi /*int **/);
// CUDEVICEGETCOUNT-NEXT: Is migrated to:
// CUDEVICEGETCOUNT-NEXT:   *pi = dpct::dev_mgr::instance().device_count();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGetName | FileCheck %s -check-prefix=CUDEVICEGETNAME
// CUDEVICEGETNAME: CUDA API:
// CUDEVICEGETNAME-NEXT:   cuDeviceGetName(pc /*char **/, i /*int*/, d /*CUdevice*/);
// CUDEVICEGETNAME-NEXT: Is migrated to:
// CUDEVICEGETNAME-NEXT:   memcpy(pc, dpct::dev_mgr::instance().get_device(d).get_info<sycl::info::device::name>().c_str(), i);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceTotalMem | FileCheck %s -check-prefix=CUDEVICETOTALMEM
// CUDEVICETOTALMEM: CUDA API:
// CUDEVICETOTALMEM-NEXT:   cuDeviceTotalMem(ps /*size_t **/, d /*CUdevice*/);
// CUDEVICETOTALMEM-NEXT: Is migrated to:
// CUDEVICETOTALMEM-NEXT:   *ps = dpct::dev_mgr::instance().get_device(d).get_device_info().get_global_mem_size();

/// Device Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceComputeCapability | FileCheck %s -check-prefix=CUDEVICECOMPUTECAPABILITY
// CUDEVICECOMPUTECAPABILITY: CUDA API:
// CUDEVICECOMPUTECAPABILITY-NEXT:   cuDeviceComputeCapability(pi1 /*int **/, pi2 /*int **/, d /*CUdevice*/);
// CUDEVICECOMPUTECAPABILITY-NEXT: Is migrated to:
// CUDEVICECOMPUTECAPABILITY-NEXT:   *pi1 = dpct::get_major_version(dpct::dev_mgr::instance().get_device(d));
// CUDEVICECOMPUTECAPABILITY-NEXT:   *pi2 = dpct::get_minor_version(dpct::dev_mgr::instance().get_device(d));

/// Primary Context Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDevicePrimaryCtxRelease | FileCheck %s -check-prefix=CUDEVICEPRIMARYCTXRELEASE
// CUDEVICEPRIMARYCTXRELEASE: CUDA API:
// CUDEVICEPRIMARYCTXRELEASE-NEXT:   cuDevicePrimaryCtxRelease(d /*CUdevice*/);
// CUDEVICEPRIMARYCTXRELEASE-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDevicePrimaryCtxRetain | FileCheck %s -check-prefix=CUDEVICEPRIMARYCTXRETAIN
// CUDEVICEPRIMARYCTXRETAIN: CUDA API:
// CUDEVICEPRIMARYCTXRETAIN-NEXT:   cuDevicePrimaryCtxRetain(pc /*CUcontext **/, d /*CUdevice*/);
// CUDEVICEPRIMARYCTXRETAIN-NEXT: Is migrated to:
// CUDEVICEPRIMARYCTXRETAIN-NEXT:   *pc = dpct::select_device(d);

/// Context Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxDestroy | FileCheck %s -check-prefix=CUCTXDESTROY
// CUCTXDESTROY: CUDA API:
// CUCTXDESTROY-NEXT:   cuCtxDestroy(c /*CUcontext*/);
// CUCTXDESTROY-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetApiVersion | FileCheck %s -check-prefix=CUCTXGETAPIVERSION
// CUCTXGETAPIVERSION: CUDA API:
// CUCTXGETAPIVERSION-NEXT:   cuCtxGetApiVersion(c /*CUcontext*/, u /*unsigned int **/);
// CUCTXGETAPIVERSION-NEXT: Is migrated to:
// CUCTXGETAPIVERSION-NEXT:   *u = dpct::get_sycl_language_version();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetCurrent | FileCheck %s -check-prefix=CUCTXGETCURRENT
// CUCTXGETCURRENT: CUDA API:
// CUCTXGETCURRENT-NEXT:   cuCtxGetCurrent(pc /*CUcontext **/);
// CUCTXGETCURRENT-NEXT: Is migrated to:
// CUCTXGETCURRENT-NEXT:   *pc = dpct::dev_mgr::instance().current_device_id();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetDevice | FileCheck %s -check-prefix=CUCTXGETDEVICE
// CUCTXGETDEVICE: CUDA API:
// CUCTXGETDEVICE-NEXT:   cuCtxGetDevice(pd /*CUdevice **/);
// CUCTXGETDEVICE-NEXT: Is migrated to:
// CUCTXGETDEVICE-NEXT:   *pd = dpct::get_current_device_id();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSetCacheConfig | FileCheck %s -check-prefix=CUCTXSETCACHECONFIG
// CUCTXSETCACHECONFIG: CUDA API:
// CUCTXSETCACHECONFIG-NEXT:   cuCtxSetCacheConfig(f /*CUfunc_cache*/);
// CUCTXSETCACHECONFIG-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSetCurrent | FileCheck %s -check-prefix=CUCTXSETCURRENT
// CUCTXSETCURRENT: CUDA API:
// CUCTXSETCURRENT-NEXT:   cuCtxSetCurrent(c /*CUcontext*/);
// CUCTXSETCURRENT-NEXT: Is migrated to:
// CUCTXSETCURRENT-NEXT:   dpct::select_device(c);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSynchronize | FileCheck %s -check-prefix=CUCTXSYNCHRONIZE
// CUCTXSYNCHRONIZE: CUDA API:
// CUCTXSYNCHRONIZE-NEXT:   cuCtxSynchronize();
// CUCTXSYNCHRONIZE-NEXT: Is migrated to:
// CUCTXSYNCHRONIZE-NEXT:   dpct::get_current_device().queues_wait_and_throw();

/// Module Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleGetFunction | FileCheck %s -check-prefix=CUMODULEGETFUNCTION
// CUMODULEGETFUNCTION: CUDA API:
// CUMODULEGETFUNCTION-NEXT:   cuModuleGetFunction(pf /*CUfunction **/, m /*CUmodule*/, pc /*const char **/);
// CUMODULEGETFUNCTION-NEXT: Is migrated to:
// CUMODULEGETFUNCTION-NEXT:   *pf = dpct::get_kernel_function(m, pc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoad | FileCheck %s -check-prefix=CUMODULELOAD
// CUMODULELOAD: CUDA API:
// CUMODULELOAD-NEXT:   cuModuleLoad(pm /*CUmodule **/, pc /*const char **/);
// CUMODULELOAD-NEXT: Is migrated to:
// CUMODULELOAD-NEXT:   *pm = dpct::load_kernel_library(pc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoadData | FileCheck %s -check-prefix=CUMODULELOADDATA
// CUMODULELOADDATA: CUDA API:
// CUMODULELOADDATA-NEXT:   cuModuleLoadData(pm /*CUmodule **/, pData /*const void **/);
// CUMODULELOADDATA-NEXT: Is migrated to:
// CUMODULELOADDATA-NEXT:    *pm = dpct::load_kernel_library_mem(pData);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoadDataEx | FileCheck %s -check-prefix=CUMODULELOADDATAEX
// CUMODULELOADDATAEX: CUDA API:
// CUMODULELOADDATAEX-NEXT:   cuModuleLoadDataEx(pm /*CUmodule **/, pData /*const void **/,
// CUMODULELOADDATAEX-NEXT:                      u /*unsigned int*/, pOpt /*CUjit_option **/,
// CUMODULELOADDATAEX-NEXT:                      pOptVal /*void ***/);
// CUMODULELOADDATAEX-NEXT: Is migrated to:
// CUMODULELOADDATAEX-NEXT:   *pm = dpct::load_kernel_library_mem(pData);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleUnload | FileCheck %s -check-prefix=CUMODULEUNLOAD
// CUMODULEUNLOAD: CUDA API:
// CUMODULEUNLOAD-NEXT:   cuModuleUnload(m /*CUmodule*/);
// CUMODULEUNLOAD-NEXT: Is migrated to:
// CUMODULEUNLOAD-NEXT:   dpct::unload_kernel_library(m);

/// Module Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleGetTexRef | FileCheck %s -check-prefix=CUMODULEGETTEXREF
// CUMODULEGETTEXREF: CUDA API:
// CUMODULEGETTEXREF-NEXT:   cuModuleGetTexRef(pt /*CUtexref **/, m /*CUmodule*/, pc /*const char **/);
// CUMODULEGETTEXREF-NEXT: Is migrated to:
// CUMODULEGETTEXREF-NEXT:   *pt = dpct::get_image_wrapper(m, pc);

/// Memory Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuArrayDestroy | FileCheck %s -check-prefix=CUARRAYDESTROY
// CUARRAYDESTROY: CUDA API:
// CUARRAYDESTROY-NEXT:   cuArrayDestroy(a /*CUarray*/);
// CUARRAYDESTROY-NEXT: Is migrated to:
// CUARRAYDESTROY-NEXT:   delete a;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAlloc | FileCheck %s -check-prefix=CUMEMALLOC
// CUMEMALLOC: CUDA API:
// CUMEMALLOC-NEXT:   cuMemAlloc(pd /*CUdeviceptr **/, s /*size_t*/);
// CUMEMALLOC-NEXT: Is migrated to:
// CUMEMALLOC-NEXT:   *pd = (dpct::device_ptr)sycl::malloc_device(s, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAllocHost | FileCheck %s -check-prefix=CUMEMALLOCHOST
// CUMEMALLOCHOST: CUDA API:
// CUMEMALLOCHOST-NEXT:   cuMemAllocHost(pHost /*void ***/, s /*size_t*/);
// CUMEMALLOCHOST-NEXT: Is migrated to:
// CUMEMALLOCHOST-NEXT:   *pHost = (void *)sycl::malloc_host(s, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAllocManaged | FileCheck %s -check-prefix=CUMEMALLOCMANAGED
// CUMEMALLOCMANAGED: CUDA API:
// CUMEMALLOCMANAGED-NEXT:   cuMemAllocManaged(pd /*CUdeviceptr **/, s /*size_t*/, u /*unsigned int*/);
// CUMEMALLOCMANAGED-NEXT: Is migrated to:
// CUMEMALLOCMANAGED-NEXT:   *pd = (dpct::device_ptr)sycl::malloc_shared(s, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAllocPitch | FileCheck %s -check-prefix=CUMEMALLOCPITCH
// CUMEMALLOCPITCH: CUDA API:
// CUMEMALLOCPITCH-NEXT:   cuMemAllocPitch(pd /*CUdeviceptr **/, ps /*size_t **/, s1 /*size_t*/,
// CUMEMALLOCPITCH-NEXT:                   s2 /*size_t*/, u /*unsigned int*/);
// CUMEMALLOCPITCH-NEXT: Is migrated to:
// CUMEMALLOCPITCH-NEXT:   *pd = (dpct::device_ptr)dpct::dpct_malloc(*ps, s1, s2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemFree | FileCheck %s -check-prefix=CUMEMFREE
// CUMEMFREE: CUDA API:
// CUMEMFREE-NEXT:   cuMemFree(d /*CUdeviceptr*/);
// CUMEMFREE-NEXT: Is migrated to:
// CUMEMFREE-NEXT:   sycl::free(d, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemFreeHost | FileCheck %s -check-prefix=CUMEMFREEHOST
// CUMEMFREEHOST: CUDA API:
// CUMEMFREEHOST-NEXT:   cuMemFreeHost(pHost /*void **/);
// CUMEMFREEHOST-NEXT: Is migrated to:
// CUMEMFREEHOST-NEXT:   sycl::free(pHost, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemGetInfo | FileCheck %s -check-prefix=CUMEMGETINFO
// CUMEMGETINFO: CUDA API:
// CUMEMGETINFO-NEXT:   cuMemGetInfo(ps1 /*size_t **/, ps2 /*size_t **/);
// CUMEMGETINFO-NEXT: Is migrated to:
// CUMEMGETINFO-NEXT:   dpct::get_current_device().get_memory_info(*ps1, *ps2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemHostAlloc | FileCheck %s -check-prefix=CUMEMHOSTALLOC
// CUMEMHOSTALLOC: CUDA API:
// CUMEMHOSTALLOC-NEXT:   cuMemHostAlloc(pHost /*void ***/, s /*size_t*/, u /*unsigned int*/);
// CUMEMHOSTALLOC-NEXT: Is migrated to:
// CUMEMHOSTALLOC-NEXT:   *pHost = (void *)sycl::malloc_host(s, dpct::get_in_order_queue());

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemHostGetDevicePointer | FileCheck %s -check-prefix=CUMEMHOSTGETDEVICEPOINTER
// CUMEMHOSTGETDEVICEPOINTER: CUDA API:
// CUMEMHOSTGETDEVICEPOINTER-NEXT:   cuMemHostGetDevicePointer(pDev /*CUdeviceptr **/, pHost /*void **/,
// CUMEMHOSTGETDEVICEPOINTER-NEXT:                             u /*unsigned int*/);
// CUMEMHOSTGETDEVICEPOINTER-NEXT: Is migrated to:
// CUMEMHOSTGETDEVICEPOINTER-NEXT:   *pDev = (dpct::device_ptr)pHost;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemHostGetFlags | FileCheck %s -check-prefix=CUMEMHOSTGETFLAGS
// CUMEMHOSTGETFLAGS: CUDA API:
// CUMEMHOSTGETFLAGS-NEXT:   cuMemHostGetFlags(pu /*unsigned int **/, pHost /*void **/);
// CUMEMHOSTGETFLAGS-NEXT: Is migrated to:
// CUMEMHOSTGETFLAGS-NEXT:   *pu = 0;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemHostRegister | FileCheck %s -check-prefix=CUMEMHOSTREGISTER
// CUMEMHOSTREGISTER: CUDA API:
// CUMEMHOSTREGISTER-NEXT:   cuMemHostRegister(pHost /*void **/, s /*size_t*/, u /*unsigned int*/);
// CUMEMHOSTREGISTER-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemHostUnregister | FileCheck %s -check-prefix=CUMEMHOSTUNREGISTER
// CUMEMHOSTUNREGISTER: CUDA API:
// CUMEMHOSTUNREGISTER-NEXT:   cuMemHostUnregister(pHost /*void **/);
// CUMEMHOSTUNREGISTER-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpy | FileCheck %s -check-prefix=CUMEMCPY
// CUMEMCPY: CUDA API:
// CUMEMCPY-NEXT:   cuMemcpy(d1 /*CUdeviceptr*/, d2 /*CUdeviceptr*/, s /*size_t*/);
// CUMEMCPY-NEXT: Is migrated to:
// CUMEMCPY-NEXT:   dpct::get_in_order_queue().memcpy(d1, d2, s).wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyAsync | FileCheck %s -check-prefix=CUMEMCPYASYNC
// CUMEMCPYASYNC: CUDA API:
// CUMEMCPYASYNC-NEXT:   CUstream cs;
// CUMEMCPYASYNC-NEXT:   cuMemcpyAsync(d1 /*CUdeviceptr*/, d2 /*CUdeviceptr*/, s /*size_t*/,
// CUMEMCPYASYNC-NEXT:                 cs /*CUstream*/);
// CUMEMCPYASYNC-NEXT: Is migrated to:
// CUMEMCPYASYNC-NEXT:   dpct::queue_ptr cs;
// CUMEMCPYASYNC-NEXT:   cs->memcpy(d1, d2, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyAtoA | FileCheck %s -check-prefix=CUMEMCPYATOA
// CUMEMCPYATOA: CUDA API:
// CUMEMCPYATOA-NEXT:   CUarray a1;
// CUMEMCPYATOA-NEXT:   CUarray a2;
// CUMEMCPYATOA-NEXT:   cuMemcpyAtoA(a1 /*CUarray*/, s1 /*size_t*/, a2 /*CUarray*/, s2 /*size_t*/,
// CUMEMCPYATOA-NEXT:                s3 /*size_t*/);
// CUMEMCPYATOA-NEXT: Is migrated to:
// CUMEMCPYATOA-NEXT:   dpct::image_matrix_p a1;
// CUMEMCPYATOA-NEXT:   dpct::image_matrix_p a2;
// CUMEMCPYATOA-NEXT:   dpct::dpct_memcpy((char *)(a1->to_pitched_data().get_data_ptr()) + s1, (char *)(a2->to_pitched_data().get_data_ptr()) + s2, s3);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyAtoD | FileCheck %s -check-prefix=CUMEMCPYATOD
// CUMEMCPYATOD: CUDA API:
// CUMEMCPYATOD-NEXT:   CUarray a;
// CUMEMCPYATOD-NEXT:   cuMemcpyAtoD(d /*CUdeviceptr*/, a /*CUarray*/, s1 /*size_t*/, s2 /*size_t*/);
// CUMEMCPYATOD-NEXT: Is migrated to:
// CUMEMCPYATOD-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYATOD-NEXT:   dpct::dpct_memcpy(d, (char *)(a->to_pitched_data().get_data_ptr()) + s1, s2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyAtoH | FileCheck %s -check-prefix=CUMEMCPYATOH
// CUMEMCPYATOH: CUDA API:
// CUMEMCPYATOH-NEXT:   CUarray a;
// CUMEMCPYATOH-NEXT:   cuMemcpyAtoH(pHost /*void **/, a /*CUarray*/, s1 /*size_t*/, s2 /*size_t*/);
// CUMEMCPYATOH-NEXT: Is migrated to:
// CUMEMCPYATOH-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYATOH-NEXT:   dpct::dpct_memcpy(pHost, (char *)(a->to_pitched_data().get_data_ptr()) + s1, s2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyAtoHAsync | FileCheck %s -check-prefix=CUMEMCPYATOHASYNC
// CUMEMCPYATOHASYNC: CUDA API:
// CUMEMCPYATOHASYNC-NEXT:   CUarray a;
// CUMEMCPYATOHASYNC-NEXT:   cuMemcpyAtoHAsync(pHost /*void **/, a /*CUarray*/, s1 /*size_t*/,
// CUMEMCPYATOHASYNC-NEXT:                     s2 /*size_t*/, s /*CUstream*/);
// CUMEMCPYATOHASYNC-NEXT: Is migrated to:
// CUMEMCPYATOHASYNC-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYATOHASYNC-NEXT:   dpct::async_dpct_memcpy(pHost, (char *)(a->to_pitched_data().get_data_ptr()) + s1, s2, dpct::automatic, *s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyDtoA | FileCheck %s -check-prefix=CUMEMCPYDTOA
// CUMEMCPYDTOA: CUDA API:
// CUMEMCPYDTOA-NEXT:   CUarray a;
// CUMEMCPYDTOA-NEXT:   cuMemcpyDtoA(a /*CUarray*/, s1 /*size_t*/, d /*CUdeviceptr*/, s2 /*size_t*/);
// CUMEMCPYDTOA-NEXT: Is migrated to:
// CUMEMCPYDTOA-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYDTOA-NEXT:   dpct::dpct_memcpy((char *)(a->to_pitched_data().get_data_ptr()) + s1, d, s2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyDtoD | FileCheck %s -check-prefix=CUMEMCPYDTOD
// CUMEMCPYDTOD: CUDA API:
// CUMEMCPYDTOD-NEXT:   cuMemcpyDtoD(pd1 /*CUdeviceptr*/, pd2 /*CUdeviceptr*/, s /*size_t*/);
// CUMEMCPYDTOD-NEXT: Is migrated to:
// CUMEMCPYDTOD-NEXT:   dpct::get_in_order_queue().memcpy(pd1, pd2, s).wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyDtoDAsync | FileCheck %s -check-prefix=CUMEMCPYDTODASYNC
// CUMEMCPYDTODASYNC: CUDA API:
// CUMEMCPYDTODASYNC-NEXT:   CUstream cs;
// CUMEMCPYDTODASYNC-NEXT:   cuMemcpyDtoDAsync(pd1 /*CUdeviceptr*/, pd2 /*CUdeviceptr*/, s /*size_t*/,
// CUMEMCPYDTODASYNC-NEXT:                     cs /*CUstream*/);
// CUMEMCPYDTODASYNC-NEXT: Is migrated to:
// CUMEMCPYDTODASYNC-NEXT:   dpct::queue_ptr cs;
// CUMEMCPYDTODASYNC-NEXT:   cs->memcpy(pd1, pd2, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyDtoH | FileCheck %s -check-prefix=CUMEMCPYDTOH
// CUMEMCPYDTOH: CUDA API:
// CUMEMCPYDTOH-NEXT:   CUdeviceptr pDev;
// CUMEMCPYDTOH-NEXT:   cuMemcpyDtoH(pHost /*void **/, pDev, s /*size_t*/);
// CUMEMCPYDTOH-NEXT: Is migrated to:
// CUMEMCPYDTOH-NEXT:   dpct::device_ptr pDev;
// CUMEMCPYDTOH-NEXT:   dpct::get_in_order_queue().memcpy(pHost /*void **/, pDev, s /*size_t*/).wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyDtoHAsync | FileCheck %s -check-prefix=CUMEMCPYDTOHASYNC
// CUMEMCPYDTOHASYNC: CUDA API:
// CUMEMCPYDTOHASYNC-NEXT:   CUdeviceptr pDev;
// CUMEMCPYDTOHASYNC-NEXT:   CUstream stream;
// CUMEMCPYDTOHASYNC-NEXT:   cuMemcpyDtoHAsync(pHost /*void **/, pDev, s /*size_t*/, stream);
// CUMEMCPYDTOHASYNC-NEXT: Is migrated to:
// CUMEMCPYDTOHASYNC-NEXT:   dpct::device_ptr pDev;
// CUMEMCPYDTOHASYNC-NEXT:   dpct::queue_ptr stream;
// CUMEMCPYDTOHASYNC-NEXT:   stream->memcpy(pHost /*void **/, pDev, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyHtoA | FileCheck %s -check-prefix=CUMEMCPYHTOA
// CUMEMCPYHTOA: CUDA API:
// CUMEMCPYHTOA-NEXT:   CUarray a;
// CUMEMCPYHTOA-NEXT:   cuMemcpyHtoA(a /*CUarray*/, s1 /*size_t*/, pHost /*const void **/,
// CUMEMCPYHTOA-NEXT:                s2 /*size_t*/);
// CUMEMCPYHTOA-NEXT: Is migrated to:
// CUMEMCPYHTOA-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYHTOA-NEXT:   dpct::dpct_memcpy((char *)(a->to_pitched_data().get_data_ptr()) + s1, pHost, s2);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyHtoAAsync | FileCheck %s -check-prefix=CUMEMCPYHTOAASYNC
// CUMEMCPYHTOAASYNC: CUDA API:
// CUMEMCPYHTOAASYNC-NEXT:   CUarray a;
// CUMEMCPYHTOAASYNC-NEXT:   cuMemcpyHtoAAsync(a /*CUarray*/, s1 /*size_t*/, pHost /*const void **/,
// CUMEMCPYHTOAASYNC-NEXT:                     s2 /*size_t*/, s /*CUstream*/);
// CUMEMCPYHTOAASYNC-NEXT: Is migrated to:
// CUMEMCPYHTOAASYNC-NEXT:   dpct::image_matrix_p a;
// CUMEMCPYHTOAASYNC-NEXT:   dpct::async_dpct_memcpy((char *)(a->to_pitched_data().get_data_ptr()) + s1, pHost, s2, dpct::automatic, *s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyHtoD | FileCheck %s -check-prefix=CUMEMCPYHTOD
// CUMEMCPYHTOD: CUDA API:
// CUMEMCPYHTOD-NEXT:   cuMemcpyHtoD(pDev /*CUdeviceptr*/, pHost /*const void **/, s /*size_t*/);
// CUMEMCPYHTOD-NEXT: Is migrated to:
// CUMEMCPYHTOD-NEXT:   dpct::get_in_order_queue().memcpy(pDev, pHost, s).wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemcpyHtoDAsync | FileCheck %s -check-prefix=CUMEMCPYHTODASYNC
// CUMEMCPYHTODASYNC: CUDA API:
// CUMEMCPYHTODASYNC-NEXT:   CUstream stream;
// CUMEMCPYHTODASYNC-NEXT:   cuMemcpyHtoDAsync(pDev /*CUdeviceptr*/, pHost /*const void **/, s /*size_t*/,
// CUMEMCPYHTODASYNC-NEXT:                     stream /*CUstream*/);
// CUMEMCPYHTODASYNC-NEXT: Is migrated to:
// CUMEMCPYHTODASYNC-NEXT:   dpct::queue_ptr stream;
// CUMEMCPYHTODASYNC-NEXT:   stream->memcpy(pDev, pHost, s);

/// Unified Addressing

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAdvise | FileCheck %s -check-prefix=CUMEMADVISE
// CUMEMADVISE: CUDA API:
// CUMEMADVISE-NEXT:   cuMemAdvise(pd /*CUdeviceptr*/, s /*size_t*/, m /*CUmem_advise*/,
// CUMEMADVISE-NEXT:               d /*CUdevice*/);
// CUMEMADVISE-NEXT: Is migrated to:
// CUMEMADVISE-NEXT:   dpct::dev_mgr::instance().get_device(d).in_order_queue().mem_advise(pd, s, m);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemPrefetchAsync | FileCheck %s -check-prefix=CUMEMPREFETCHASYNC
// CUMEMPREFETCHASYNC: CUDA API:
// CUMEMPREFETCHASYNC-NEXT:   CUstream cs;
// CUMEMPREFETCHASYNC-NEXT:   cuMemPrefetchAsync(pd /*CUdeviceptr*/, s /*size_t*/, d /*CUdevice*/,
// CUMEMPREFETCHASYNC-NEXT:                      cs /*CUstream*/);
// CUMEMPREFETCHASYNC-NEXT: Is migrated to:
// CUMEMPREFETCHASYNC-NEXT:   dpct::queue_ptr cs;
// CUMEMPREFETCHASYNC-NEXT:   cs->prefetch(pd, s);

/// Stream Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamAddCallback | FileCheck %s -check-prefix=CUSTREAMADDCALLBACK
// CUSTREAMADDCALLBACK: CUDA API:
// CUSTREAMADDCALLBACK-NEXT:   CUstream s;
// CUSTREAMADDCALLBACK-NEXT:   cuStreamAddCallback(s /*CUstream*/, sc /*CUstreamCallback*/, pData /*void **/,
// CUSTREAMADDCALLBACK-NEXT:                       u /*unsigned int*/);
// CUSTREAMADDCALLBACK-NEXT: Is migrated to:
// CUSTREAMADDCALLBACK-NEXT:   dpct::queue_ptr s;
// CUSTREAMADDCALLBACK-NEXT:   std::async([&](){s->wait(); sc(s, 0, pData);});

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamAttachMemAsync | FileCheck %s -check-prefix=CUSTREAMATTACHMEMASYNC
// CUSTREAMATTACHMEMASYNC: CUDA API:
// CUSTREAMATTACHMEMASYNC-NEXT:   cuStreamAttachMemAsync(cs /*CUstream*/, d /*CUdeviceptr*/, s /*size_t*/,
// CUSTREAMATTACHMEMASYNC-NEXT:                          u /*unsigned int*/);
// CUSTREAMATTACHMEMASYNC-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamCreate | FileCheck %s -check-prefix=CUSTREAMCREATE
// CUSTREAMCREATE: CUDA API:
// CUSTREAMCREATE-NEXT:   cuStreamCreate(ps /*CUstream **/, u /*unsigned int*/);
// CUSTREAMCREATE-NEXT: Is migrated to:
// CUSTREAMCREATE-NEXT:   *(ps) = dpct::get_current_device().create_queue();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamDestroy | FileCheck %s -check-prefix=CUSTREAMDESTROY
// CUSTREAMDESTROY: CUDA API:
// CUSTREAMDESTROY-NEXT:   cuStreamDestroy(s /*CUstream*/);
// CUSTREAMDESTROY-NEXT: Is migrated to:
// CUSTREAMDESTROY-NEXT:   dpct::get_current_device().destroy_queue(s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamSynchronize | FileCheck %s -check-prefix=CUSTREAMSYNCHRONIZE
// CUSTREAMSYNCHRONIZE: CUDA API:
// CUSTREAMSYNCHRONIZE-NEXT:   CUstream s;
// CUSTREAMSYNCHRONIZE-NEXT:   cuStreamSynchronize(s /*CUstream*/);
// CUSTREAMSYNCHRONIZE-NEXT: Is migrated to:
// CUSTREAMSYNCHRONIZE-NEXT:   dpct::queue_ptr s;
// CUSTREAMSYNCHRONIZE-NEXT:   s->wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamWaitEvent | FileCheck %s -check-prefix=CUSTREAMWAITEVENT
// CUSTREAMWAITEVENT: CUDA API:
// CUSTREAMWAITEVENT-NEXT:   CUstream s;
// CUSTREAMWAITEVENT-NEXT:   cuStreamWaitEvent(s /*CUstream*/, e /*CUevent*/, u /*unsigned int*/);
// CUSTREAMWAITEVENT-NEXT: Is migrated to:
// CUSTREAMWAITEVENT-NEXT:   dpct::queue_ptr s;
// CUSTREAMWAITEVENT-NEXT:   s->ext_oneapi_submit_barrier({*e});

/// Event Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventCreate | FileCheck %s -check-prefix=CUEVENTCREATE
// CUEVENTCREATE: CUDA API:
// CUEVENTCREATE-NEXT:   cuEventCreate(pe /*CUevent **/, u /*unsigned int*/);
// CUEVENTCREATE-NEXT: Is migrated to:
// CUEVENTCREATE-NEXT:   *pe = new sycl::event();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventDestroy | FileCheck %s -check-prefix=CUEVENTDESTROY
// CUEVENTDESTROY: CUDA API:
// CUEVENTDESTROY-NEXT:   cuEventDestroy(e /*CUevent*/);
// CUEVENTDESTROY-NEXT: Is migrated to:
// CUEVENTDESTROY-NEXT:   dpct::destroy_event(e);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventElapsedTime | FileCheck %s -check-prefix=CUEVENTELAPSEDTIME
// CUEVENTELAPSEDTIME: CUDA API:
// CUEVENTELAPSEDTIME-NEXT:   cuEventElapsedTime(pf /*float **/, e1 /*CUevent*/, e2 /*CUevent*/);
// CUEVENTELAPSEDTIME-NEXT: Is migrated to:
// CUEVENTELAPSEDTIME-NEXT:   *(pf) = std::chrono::duration<float, std::milli>(e2_ct1 - e1_ct1).count();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventQuery | FileCheck %s -check-prefix=CUEVENTQUERY
// CUEVENTQUERY: CUDA API:
// CUEVENTQUERY-NEXT:   CUevent e;
// CUEVENTQUERY-NEXT:   cuEventQuery(e /*CUevent*/);
// CUEVENTQUERY-NEXT: Is migrated to:
// CUEVENTQUERY-NEXT:   dpct::event_ptr e;
// CUEVENTQUERY-NEXT:   (int)e->get_info<sycl::info::event::command_execution_status>();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventRecord | FileCheck %s -check-prefix=CUEVENTRECORD
// CUEVENTRECORD: CUDA API:
// CUEVENTRECORD-NEXT:   CUstream s;
// CUEVENTRECORD-NEXT:   cuEventRecord(e /*CUevent*/, s /*CUstream*/);
// CUEVENTRECORD-NEXT: Is migrated to:
// CUEVENTRECORD-NEXT:   dpct::queue_ptr s;
// CUEVENTRECORD-NEXT:   ;
// CUEVENTRECORD-NEXT:   *e = s->ext_oneapi_submit_barrier();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventSynchronize | FileCheck %s -check-prefix=CUEVENTSYNCHRONIZE
// CUEVENTSYNCHRONIZE: CUDA API:
// CUEVENTSYNCHRONIZE-NEXT:   CUevent e;
// CUEVENTSYNCHRONIZE-NEXT:   cuEventSynchronize(e /*CUevent*/);
// CUEVENTSYNCHRONIZE-NEXT: Is migrated to:
// CUEVENTSYNCHRONIZE-NEXT:   dpct::event_ptr e;
// CUEVENTSYNCHRONIZE-NEXT:   e->wait_and_throw();

/// Execution Control

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuFuncGetAttribute | FileCheck %s -check-prefix=CUFUNCGETATTRIBUTE
// CUFUNCGETATTRIBUTE: CUDA API:
// CUFUNCGETATTRIBUTE-NEXT:   cuFuncGetAttribute(pi /*int **/, fa /*CUfunction_attribute*/,
// CUFUNCGETATTRIBUTE-NEXT:                      f /*CUfunction*/);
// CUFUNCGETATTRIBUTE-NEXT: Is migrated to:
// CUFUNCGETATTRIBUTE-NEXT:   *pi = dpct::get_kernel_function_info(f).fa;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuFuncSetCacheConfig | FileCheck %s -check-prefix=CUFUNCSETCACHECONFIG
// CUFUNCSETCACHECONFIG: CUDA API:
// CUFUNCSETCACHECONFIG-NEXT:   cuFuncSetCacheConfig(f /*CUfunction*/, fc /*CUfunc_cache*/);
// CUFUNCSETCACHECONFIG-NEXT: The API is Removed.

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuLaunchKernel | FileCheck %s -check-prefix=CULAUNCHKERNEL
// CULAUNCHKERNEL: CUDA API:
// CULAUNCHKERNEL-NEXT:   cuLaunchKernel(f /*CUfunction*/, u1 /*unsigned int*/, u2 /*unsigned int*/,
// CULAUNCHKERNEL-NEXT:                  u3 /*unsigned int*/, u4 /*unsigned int*/, u5 /*unsigned int*/,
// CULAUNCHKERNEL-NEXT:                  u6 /*unsigned int*/, u7 /*unsigned int*/, s /*CUstream*/,
// CULAUNCHKERNEL-NEXT:                  pParam /*void ***/, pOpt /*void ***/);
// CULAUNCHKERNEL-NEXT: Is migrated to:
// CULAUNCHKERNEL-NEXT:   dpct::invoke_kernel_function(f, *s, sycl::range<3>(u3, u2, u1), sycl::range<3>(u6, u5, u4), u7, pParam, pOpt);

/// Texture Reference Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetAddressMode | FileCheck %s -check-prefix=CUTEXREFGETADDRESSMODE
// CUTEXREFGETADDRESSMODE: CUDA API:
// CUTEXREFGETADDRESSMODE-NEXT:   CUtexref t;
// CUTEXREFGETADDRESSMODE-NEXT:   cuTexRefGetAddressMode(pa /*CUaddress_mode **/, t /*CUtexref*/, i /*int*/);
// CUTEXREFGETADDRESSMODE-NEXT: Is migrated to:
// CUTEXREFGETADDRESSMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETADDRESSMODE-NEXT:   *pa = t->get_addressing_mode();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetFilterMode | FileCheck %s -check-prefix=CUTEXREFGETFILTERMODE
// CUTEXREFGETFILTERMODE: CUDA API:
// CUTEXREFGETFILTERMODE-NEXT:   CUtexref t;
// CUTEXREFGETFILTERMODE-NEXT:   cuTexRefGetFilterMode(pf /*CUfilter_mode **/, t /*CUtexref*/);
// CUTEXREFGETFILTERMODE-NEXT: Is migrated to:
// CUTEXREFGETFILTERMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETFILTERMODE-NEXT:   *pf = t->get_filtering_mode();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetFlags | FileCheck %s -check-prefix=CUTEXREFGETFLAGS
// CUTEXREFGETFLAGS: CUDA API:
// CUTEXREFGETFLAGS-NEXT:   CUtexref t;
// CUTEXREFGETFLAGS-NEXT:   cuTexRefGetFlags(pu /*unsigned int **/, t /*CUtexref*/);
// CUTEXREFGETFLAGS-NEXT: Is migrated to:
// CUTEXREFGETFLAGS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETFLAGS-NEXT:   *pu = t->is_coordinate_normalized() << 1;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddress | FileCheck %s -check-prefix=CUTEXREFSETADDRESS
// CUTEXREFSETADDRESS: CUDA API:
// CUTEXREFSETADDRESS-NEXT:   CUtexref t;
// CUTEXREFSETADDRESS-NEXT:   cuTexRefSetAddress(ps /*size_t **/, t /*CUtexref*/, d /*CUdeviceptr*/,
// CUTEXREFSETADDRESS-NEXT:                      s /*size_t*/);
// CUTEXREFSETADDRESS-NEXT: Is migrated to:
// CUTEXREFSETADDRESS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESS-NEXT:   t->attach(d, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddress2D | FileCheck %s -check-prefix=CUTEXREFSETADDRESS2D
// CUTEXREFSETADDRESS2D: CUDA API:
// CUTEXREFSETADDRESS2D-NEXT:   CUtexref t;
// CUTEXREFSETADDRESS2D-NEXT:   cuTexRefSetAddress2D(t /*CUtexref*/, pa /*size_t **/, d /*CUdeviceptr*/,
// CUTEXREFSETADDRESS2D-NEXT:                        s /*size_t*/);
// CUTEXREFSETADDRESS2D-NEXT: Is migrated to:
// CUTEXREFSETADDRESS2D-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESS2D-NEXT:   t->attach(d, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddressMode | FileCheck %s -check-prefix=CUTEXREFSETADDRESSMODE
// CUTEXREFSETADDRESSMODE: CUDA API:
// CUTEXREFSETADDRESSMODE-NEXT:   CUtexref t;
// CUTEXREFSETADDRESSMODE-NEXT:   cuTexRefSetAddressMode(t /*CUtexref*/, i /*int **/, a /*CUaddress_mode*/);
// CUTEXREFSETADDRESSMODE-NEXT: Is migrated to:
// CUTEXREFSETADDRESSMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESSMODE-NEXT:   t->set(a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetArray | FileCheck %s -check-prefix=CUTEXREFSETARRAY
// CUTEXREFSETARRAY: CUDA API:
// CUTEXREFSETARRAY-NEXT:   CUtexref t;
// CUTEXREFSETARRAY-NEXT:   cuTexRefSetArray(t /*CUtexref*/, a /*CUarray*/, u /*unsigned int*/);
// CUTEXREFSETARRAY-NEXT: Is migrated to:
// CUTEXREFSETARRAY-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETARRAY-NEXT:   t->attach(dpct::image_data(a));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFilterMode | FileCheck %s -check-prefix=CUTEXREFSETFILTERMODE
// CUTEXREFSETFILTERMODE: CUDA API:
// CUTEXREFSETFILTERMODE-NEXT:   CUtexref t;
// CUTEXREFSETFILTERMODE-NEXT:   cuTexRefSetFilterMode(t /*CUtexref*/, f /*CUfilter_mode*/);
// CUTEXREFSETFILTERMODE-NEXT: Is migrated to:
// CUTEXREFSETFILTERMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFILTERMODE-NEXT:   t->set(f);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFlags | FileCheck %s -check-prefix=CUTEXREFSETFLAGS
// CUTEXREFSETFLAGS: CUDA API:
// CUTEXREFSETFLAGS-NEXT:   CUtexref t;
// CUTEXREFSETFLAGS-NEXT:   cuTexRefSetFlags(t /*CUtexref*/, u /*unsigned int*/);
// CUTEXREFSETFLAGS-NEXT: Is migrated to:
// CUTEXREFSETFLAGS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFLAGS-NEXT:   t->set_coordinate_normalization_mode(u & 0x02);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFormat | FileCheck %s -check-prefix=CUTEXREFSETFORMAT
// CUTEXREFSETFORMAT: CUDA API:
// CUTEXREFSETFORMAT-NEXT:   CUtexref t;
// CUTEXREFSETFORMAT-NEXT:   cuTexRefSetFormat(t /*CUtexref*/, a /*CUarray_format*/, i /*int*/);
// CUTEXREFSETFORMAT-NEXT: Is migrated to:
// CUTEXREFSETFORMAT-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFORMAT-NEXT:   t->set_channel_type(a);
// CUTEXREFSETFORMAT-NEXT:   t->set_channel_num(i);

/// Texture Object Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectCreate | FileCheck %s -check-prefix=CUTEXOBJECTCREATE
// CUTEXOBJECTCREATE: CUDA API:
// CUTEXOBJECTCREATE-NEXT:   cuTexObjectCreate(pt /*CUtexObject **/, pr /*const CUDA_RESOURCE_DESC **/,
// CUTEXOBJECTCREATE-NEXT:                     ptd /*const CUDA_TEXTURE_DESC **/,
// CUTEXOBJECTCREATE-NEXT:                     prv /*const CUDA_RESOURCE_VIEW_DESC **/);
// CUTEXOBJECTCREATE-NEXT: Is migrated to:
// CUTEXOBJECTCREATE-NEXT:   *pt = dpct::create_image_wrapper(*pr, *ptd);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectDestroy | FileCheck %s -check-prefix=CUTEXOBJECTDESTROY
// CUTEXOBJECTDESTROY: CUDA API:
// CUTEXOBJECTDESTROY-NEXT:   cuTexObjectDestroy(t /*CUtexObject*/);
// CUTEXOBJECTDESTROY-NEXT: Is migrated to:
// CUTEXOBJECTDESTROY-NEXT:   delete t;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectGetResourceDesc | FileCheck %s -check-prefix=CUTEXOBJECTGETRESOURCEDESC
// CUTEXOBJECTGETRESOURCEDESC: CUDA API:
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   CUtexObject t;
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   cuTexObjectGetResourceDesc(pr /*CUDA_RESOURCE_DESC **/, t /*CUtexObject*/);
// CUTEXOBJECTGETRESOURCEDESC-NEXT: Is migrated to:
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   *pr = t->get_data();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectGetTextureDesc | FileCheck %s -check-prefix=CUTEXOBJECTGETTEXTUREDESC
// CUTEXOBJECTGETTEXTUREDESC: CUDA API:
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   CUtexObject t;
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   cuTexObjectGetTextureDesc(pt /*CUDA_TEXTURE_DESC **/, t /*CUtexObject*/);
// CUTEXOBJECTGETTEXTUREDESC-NEXT: Is migrated to:
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   *pt = t->get_sampling_info();
