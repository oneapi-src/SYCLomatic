
#include <hip/hip_runtime.h>
/// Error Handling

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuGetErrorString | FileCheck %s -check-prefix=CUGETERRORSTRING
// CUGETERRORSTRING: CUDA API:
// CUGETERRORSTRING-NEXT:   cuGetErrorString(r /*CUresult*/, ppc /*const char ***/);
// CUGETERRORSTRING-NEXT: Is migrated to:
// CUGETERRORSTRING-NEXT:   /*
// CUGETERRORSTRING-NEXT:   DPCT1009:0: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced by a placeholder string. You need to rewrite this code.
// CUGETERRORSTRING-NEXT:   */
// CUGETERRORSTRING-NEXT:   *ppc = "<Placeholder string>";

/// Initialization

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuInit | FileCheck %s -check-prefix=CUINIT
// CUINIT: CUDA API:
// CUINIT-NEXT:   cuInit(u /*unsigned int*/);
// CUINIT-NEXT: The API is Removed.
// CUINIT-EMPTY:

/// Version Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDriverGetVersion | FileCheck %s -check-prefix=CUDRIVERGETVERSION
// CUDRIVERGETVERSION: CUDA API:
// CUDRIVERGETVERSION-NEXT:   cuDriverGetVersion(pi /*int **/);
// CUDRIVERGETVERSION-NEXT: Is migrated to:
// CUDRIVERGETVERSION-NEXT:   *pi = std::stoi(dpct::get_current_device().get_info<sycl::info::device::version>());

/// Device Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGet | FileCheck %s -check-prefix=CUDEVICEGET
// CUDEVICEGET: CUDA API:
// CUDEVICEGET-NEXT:   cuDeviceGet(pd /*CUdevice **/, i /*int*/);
// CUDEVICEGET-NEXT: Is migrated to:
// CUDEVICEGET-NEXT:   *pd = i;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGetAttribute | FileCheck %s -check-prefix=CUDEVICEGETATTRIBUTE
// CUDEVICEGETATTRIBUTE: CUDA API:
// CUDEVICEGETATTRIBUTE:   /* 1 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_MAX_THREADS_PER_BLOCK, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 2 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_MAX_BLOCK_DIM_X, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 3 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_MAX_BLOCK_DIM_Y, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 4 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_MAX_BLOCK_DIM_Z, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 5 */ cuDeviceGetAttribute(pi /*int **/,
// CUDEVICEGETATTRIBUTE:                                CU_DEVICE_ATTRIBUTE_MAX_SHARED_MEMORY_PER_BLOCK,
// CUDEVICEGETATTRIBUTE:                                d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 6 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_TOTAL_CONSTANT_MEMORY, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 7 */ cuDeviceGetAttribute(pi /*int **/, CU_DEVICE_ATTRIBUTE_WARP_SIZE,
// CUDEVICEGETATTRIBUTE:                                d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 8 */ cuDeviceGetAttribute(pi /*int **/,
// CUDEVICEGETATTRIBUTE:                                CU_DEVICE_ATTRIBUTE_MAX_REGISTERS_PER_BLOCK,
// CUDEVICEGETATTRIBUTE:                                d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 9 */ cuDeviceGetAttribute(pi /*int **/, CU_DEVICE_ATTRIBUTE_CLOCK_RATE,
// CUDEVICEGETATTRIBUTE:                                d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 10 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_TEXTURE_ALIGNMENT, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 11 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_MULTIPROCESSOR_COUNT, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 12 */ cuDeviceGetAttribute(pi /*int **/, CU_DEVICE_ATTRIBUTE_INTEGRATED,
// CUDEVICEGETATTRIBUTE:                                 d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 13 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_CAN_MAP_HOST_MEMORY, d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 14 */ cuDeviceGetAttribute(pi /*int **/,
// CUDEVICEGETATTRIBUTE:                                 CU_DEVICE_ATTRIBUTE_COMPUTE_CAPABILITY_MAJOR,
// CUDEVICEGETATTRIBUTE:                                 d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 15 */ cuDeviceGetAttribute(pi /*int **/,
// CUDEVICEGETATTRIBUTE:                                 CU_DEVICE_ATTRIBUTE_COMPUTE_CAPABILITY_MINOR,
// CUDEVICEGETATTRIBUTE:                                 d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE:   /* 16 */ cuDeviceGetAttribute(
// CUDEVICEGETATTRIBUTE:       pi /*int **/, CU_DEVICE_ATTRIBUTE_HOST_NATIVE_ATOMIC_SUPPORTED,
// CUDEVICEGETATTRIBUTE:       d /*CUdevice*/);
// CUDEVICEGETATTRIBUTE-NEXT: Is migrated to:
// CUDEVICEGETATTRIBUTE-NEXT:  /* 1 */ *pi = dpct::get_device(d).get_max_work_group_size();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 2 */ *pi = dpct::get_device(d).get_device_info().get_max_work_item_sizes().get(0);
// CUDEVICEGETATTRIBUTE-NEXT:  /* 3 */ *pi = dpct::get_device(d).get_device_info().get_max_work_item_sizes().get(1);
// CUDEVICEGETATTRIBUTE-NEXT:  /* 4 */ *pi = dpct::get_device(d).get_device_info().get_max_work_item_sizes().get(2);
// CUDEVICEGETATTRIBUTE-NEXT:  /* 5 */ *pi = dpct::get_device(d).get_device_info().get_local_mem_size();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 6 */ *pi = dpct::get_device(d).get_global_mem_size();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 7 */ *pi = dpct::get_device(d).get_max_sub_group_size();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 8 */ *pi = dpct::get_device(d).get_max_register_size_per_work_group();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 9 */ *pi = dpct::get_device(d).get_max_clock_frequency();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 10 */ *pi = dpct::get_device(d).get_mem_base_addr_align_in_bytes();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 11 */ *pi = dpct::get_device(d).get_max_compute_units();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 12 */ *pi = dpct::get_device(d).get_integrated();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 13 */ *pi = dpct::get_device(d).has(sycl::aspect::usm_host_allocations);
// CUDEVICEGETATTRIBUTE-NEXT:  /* 14 */ *pi = dpct::get_device(d).get_major_version();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 15 */ *pi = dpct::get_device(d).get_minor_version();
// CUDEVICEGETATTRIBUTE-NEXT:  /* 16 */ *pi = dpct::get_device(d).is_native_atomic_supported();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGetCount | FileCheck %s -check-prefix=CUDEVICEGETCOUNT
// CUDEVICEGETCOUNT: CUDA API:
// CUDEVICEGETCOUNT-NEXT:   cuDeviceGetCount(pi /*int **/);
// CUDEVICEGETCOUNT-NEXT: Is migrated to:
// CUDEVICEGETCOUNT-NEXT:   *pi = dpct::device_count();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceGetName | FileCheck %s -check-prefix=CUDEVICEGETNAME
// CUDEVICEGETNAME: CUDA API:
// CUDEVICEGETNAME-NEXT:   cuDeviceGetName(pc /*char **/, i /*int*/, d /*CUdevice*/);
// CUDEVICEGETNAME-NEXT: Is migrated to:
// CUDEVICEGETNAME-NEXT:   memcpy(pc, dpct::get_device(d).get_info<sycl::info::device::name>().c_str(), i);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceTotalMem | FileCheck %s -check-prefix=CUDEVICETOTALMEM
// CUDEVICETOTALMEM: CUDA API:
// CUDEVICETOTALMEM-NEXT:   cuDeviceTotalMem(ps /*size_t **/, d /*CUdevice*/);
// CUDEVICETOTALMEM-NEXT: Is migrated to:
// CUDEVICETOTALMEM-NEXT:   *ps = dpct::get_device(d).get_device_info().get_global_mem_size();

/// Device Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceComputeCapability | FileCheck %s -check-prefix=CUDEVICECOMPUTECAPABILITY
// CUDEVICECOMPUTECAPABILITY: CUDA API:
// CUDEVICECOMPUTECAPABILITY-NEXT:   cuDeviceComputeCapability(pi1 /*int **/, pi2 /*int **/, d /*CUdevice*/);
// CUDEVICECOMPUTECAPABILITY-NEXT: Is migrated to:
// CUDEVICECOMPUTECAPABILITY-NEXT:   *pi1 = dpct::get_major_version(dpct::get_device(d));
// CUDEVICECOMPUTECAPABILITY-NEXT:   *pi2 = dpct::get_minor_version(dpct::get_device(d));

/// Primary Context Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDevicePrimaryCtxRelease | FileCheck %s -check-prefix=CUDEVICEPRIMARYCTXRELEASE
// CUDEVICEPRIMARYCTXRELEASE: CUDA API:
// CUDEVICEPRIMARYCTXRELEASE-NEXT:   cuDevicePrimaryCtxRelease(d /*CUdevice*/);
// CUDEVICEPRIMARYCTXRELEASE-NEXT: The API is Removed.
// CUDEVICEPRIMARYCTXRELEASE-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDevicePrimaryCtxRetain | FileCheck %s -check-prefix=CUDEVICEPRIMARYCTXRETAIN
// CUDEVICEPRIMARYCTXRETAIN: CUDA API:
// CUDEVICEPRIMARYCTXRETAIN-NEXT:   cuDevicePrimaryCtxRetain(pc /*CUcontext **/, d /*CUdevice*/);
// CUDEVICEPRIMARYCTXRETAIN-NEXT: Is migrated to:
// CUDEVICEPRIMARYCTXRETAIN-NEXT:   *pc = dpct::select_device(d);

/// Context Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxCreate | FileCheck %s -check-prefix=CUCTXCREATE
// CUCTXCREATE: CUDA API:
// CUCTXCREATE-NEXT:   cuCtxCreate(pc /*CUcontext **/, u /*unsigned*/, d /*CUdevice*/);
// CUCTXCREATE-NEXT: Is migrated to:
// CUCTXCREATE-NEXT:   *pc = dpct::push_device_for_curr_thread(d);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxDestroy | FileCheck %s -check-prefix=CUCTXDESTROY
// CUCTXDESTROY: CUDA API:
// CUCTXDESTROY-NEXT:   cuCtxDestroy(c /*CUcontext*/);
// CUCTXDESTROY-NEXT: The API is Removed.
// CUCTXDESTROY-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetApiVersion | FileCheck %s -check-prefix=CUCTXGETAPIVERSION
// CUCTXGETAPIVERSION: CUDA API:
// CUCTXGETAPIVERSION-NEXT:   cuCtxGetApiVersion(c /*CUcontext*/, u /*unsigned int **/);
// CUCTXGETAPIVERSION-NEXT: Is migrated to:
// CUCTXGETAPIVERSION-NEXT:   *u = dpct::get_sycl_language_version();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetCurrent | FileCheck %s -check-prefix=CUCTXGETCURRENT
// CUCTXGETCURRENT: CUDA API:
// CUCTXGETCURRENT-NEXT:   cuCtxGetCurrent(pc /*CUcontext **/);
// CUCTXGETCURRENT-NEXT: Is migrated to:
// CUCTXGETCURRENT-NEXT:   *pc = dpct::get_current_device_id();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetDevice | FileCheck %s -check-prefix=CUCTXGETDEVICE
// CUCTXGETDEVICE: CUDA API:
// CUCTXGETDEVICE-NEXT:   cuCtxGetDevice(pd /*CUdevice **/);
// CUCTXGETDEVICE-NEXT: Is migrated to:
// CUCTXGETDEVICE-NEXT:   *pd = dpct::get_current_device_id();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxPushCurrent | FileCheck %s -check-prefix=CUCTXPUSHCURRENT
// CUCTXPUSHCURRENT: CUDA API:
// CUCTXPUSHCURRENT-NEXT:   cuCtxPushCurrent(c /*CUcontext*/);
// CUCTXPUSHCURRENT-NEXT: Is migrated to:
// CUCTXPUSHCURRENT-NEXT:   dpct::push_device_for_curr_thread(c);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxPopCurrent | FileCheck %s -check-prefix=CUCTXPOPCURRENT
// CUCTXPOPCURRENT: CUDA API:
// CUCTXPOPCURRENT-NEXT:   cuCtxPopCurrent(&c /*CUcontext **/);
// CUCTXPOPCURRENT-NEXT: Is migrated to:
// CUCTXPOPCURRENT-NEXT:   c = dpct::pop_device_for_curr_thread();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxGetLimit | FileCheck %s -check-prefix=CUCTXGETLIMIT
// CUCTXGETLIMIT: CUDA API:
// CUCTXGETLIMIT-NEXT:   cuCtxGetLimit(ps /*size_t **/, CU_LIMIT_PRINTF_FIFO_SIZE);
// CUCTXGETLIMIT-NEXT: Is migrated to:
// CUCTXGETLIMIT-NEXT:   *ps = INT_MAX;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSetCacheConfig | FileCheck %s -check-prefix=CUCTXSETCACHECONFIG
// CUCTXSETCACHECONFIG: CUDA API:
// CUCTXSETCACHECONFIG-NEXT:   cuCtxSetCacheConfig(f /*CUfunc_cache*/);
// CUCTXSETCACHECONFIG-NEXT: The API is Removed.
// CUCTXSETCACHECONFIG-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSetCurrent | FileCheck %s -check-prefix=CUCTXSETCURRENT
// CUCTXSETCURRENT: CUDA API:
// CUCTXSETCURRENT-NEXT:   cuCtxSetCurrent(c /*CUcontext*/);
// CUCTXSETCURRENT-NEXT: Is migrated to:
// CUCTXSETCURRENT-NEXT:   dpct::select_device(c);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSetLimit | FileCheck %s -check-prefix=CUCTXSETLIMIT
// CUCTXSETLIMIT: CUDA API:
// CUCTXSETLIMIT-NEXT:   cuCtxSetLimit(l /*CUlimit*/, s /*size_t*/);
// CUCTXSETLIMIT-NEXT: The API is Removed.
// CUCTXSETLIMIT-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxSynchronize | FileCheck %s -check-prefix=CUCTXSYNCHRONIZE
// CUCTXSYNCHRONIZE: CUDA API:
// CUCTXSYNCHRONIZE-NEXT:   cuCtxSynchronize();
// CUCTXSYNCHRONIZE-NEXT: Is migrated to:
// CUCTXSYNCHRONIZE-NEXT:   dpct::get_current_device().queues_wait_and_throw();

/// Module Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleGetFunction | FileCheck %s -check-prefix=CUMODULEGETFUNCTION
// CUMODULEGETFUNCTION: CUDA API:
// CUMODULEGETFUNCTION-NEXT:   cuModuleGetFunction(pf /*CUfunction **/, m /*CUmodule*/, pc /*const char **/);
// CUMODULEGETFUNCTION-NEXT: Is migrated to:
// CUMODULEGETFUNCTION-NEXT:   *pf = dpct::get_kernel_function(m, pc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoad | FileCheck %s -check-prefix=CUMODULELOAD
// CUMODULELOAD: CUDA API:
// CUMODULELOAD-NEXT:   cuModuleLoad(pm /*CUmodule **/, pc /*const char **/);
// CUMODULELOAD-NEXT: Is migrated to:
// CUMODULELOAD-NEXT:   *pm = dpct::load_kernel_library(pc);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoadData | FileCheck %s -check-prefix=CUMODULELOADDATA
// CUMODULELOADDATA: CUDA API:
// CUMODULELOADDATA-NEXT:   cuModuleLoadData(pm /*CUmodule **/, pData /*const void **/);
// CUMODULELOADDATA-NEXT: Is migrated to:
// CUMODULELOADDATA-NEXT:    *pm = dpct::load_kernel_library_mem(pData);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleLoadDataEx | FileCheck %s -check-prefix=CUMODULELOADDATAEX
// CUMODULELOADDATAEX: CUDA API:
// CUMODULELOADDATAEX-NEXT:   cuModuleLoadDataEx(pm /*CUmodule **/, pData /*const void **/,
// CUMODULELOADDATAEX-NEXT:                      u /*unsigned int*/, pOpt /*CUjit_option **/,
// CUMODULELOADDATAEX-NEXT:                      pOptVal /*void ***/);
// CUMODULELOADDATAEX-NEXT: Is migrated to:
// CUMODULELOADDATAEX-NEXT:   *pm = dpct::load_kernel_library_mem(pData);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleUnload | FileCheck %s -check-prefix=CUMODULEUNLOAD
// CUMODULEUNLOAD: CUDA API:
// CUMODULEUNLOAD-NEXT:   cuModuleUnload(m /*CUmodule*/);
// CUMODULEUNLOAD-NEXT: Is migrated to:
// CUMODULEUNLOAD-NEXT:   dpct::unload_kernel_library(m);

/// Module Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuModuleGetTexRef | FileCheck %s -check-prefix=CUMODULEGETTEXREF
// CUMODULEGETTEXREF: CUDA API:
// CUMODULEGETTEXREF-NEXT:   cuModuleGetTexRef(pt /*CUtexref **/, m /*CUmodule*/, pc /*const char **/);
// CUMODULEGETTEXREF-NEXT: Is migrated to:
// CUMODULEGETTEXREF-NEXT:   *pt = dpct::get_image_wrapper(m, pc);

/// Unified Addressing

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemAdvise | FileCheck %s -check-prefix=CUMEMADVISE
// CUMEMADVISE: CUDA API:
// CUMEMADVISE-NEXT:   cuMemAdvise(pd /*CUdeviceptr*/, s /*size_t*/, m /*CUmem_advise*/,
// CUMEMADVISE-NEXT:               d /*CUdevice*/);
// CUMEMADVISE-NEXT: Is migrated to:
// CUMEMADVISE-NEXT:   dpct::get_device(d).in_order_queue().mem_advise(pd, s, m);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuMemPrefetchAsync | FileCheck %s -check-prefix=CUMEMPREFETCHASYNC
// CUMEMPREFETCHASYNC: CUDA API:
// CUMEMPREFETCHASYNC-NEXT:   CUstream cs;
// CUMEMPREFETCHASYNC-NEXT:   cuMemPrefetchAsync(pd /*CUdeviceptr*/, s /*size_t*/, d /*CUdevice*/,
// CUMEMPREFETCHASYNC-NEXT:                      cs /*CUstream*/);
// CUMEMPREFETCHASYNC-NEXT: Is migrated to:
// CUMEMPREFETCHASYNC-NEXT:   dpct::queue_ptr cs;
// CUMEMPREFETCHASYNC-NEXT:   cs->prefetch(pd, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuPointerGetAttributes | FileCheck %s -check-prefix=CUPOINTERGETATTRIBUTES
// CUPOINTERGETATTRIBUTES: CUDA API:
// CUPOINTERGETATTRIBUTES-NEXT:   cuPointerGetAttributes(numAttr /*unsigned int*/,
// CUPOINTERGETATTRIBUTES-NEXT:                            attr /*CUpointer_attribute **/, data /*void ***/,
// CUPOINTERGETATTRIBUTES-NEXT:                            ptr /*CUdeviceptr*/);
// CUPOINTERGETATTRIBUTES-NEXT: Is migrated to:
// CUPOINTERGETATTRIBUTES-NEXT:   dpct::pointer_attributes::get(numAttr, attr, data, ptr);

/// Stream Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamAddCallback | FileCheck %s -check-prefix=CUSTREAMADDCALLBACK
// CUSTREAMADDCALLBACK: CUDA API:
// CUSTREAMADDCALLBACK-NEXT:   CUstream s;
// CUSTREAMADDCALLBACK-NEXT:   cuStreamAddCallback(s /*CUstream*/, sc /*CUstreamCallback*/, pData /*void **/,
// CUSTREAMADDCALLBACK-NEXT:                       u /*unsigned int*/);
// CUSTREAMADDCALLBACK-NEXT: Is migrated to:
// CUSTREAMADDCALLBACK-NEXT:   dpct::queue_ptr s;
// CUSTREAMADDCALLBACK-NEXT:   std::async([&]() { s->wait(); sc(s, 0, pData); });

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamAttachMemAsync | FileCheck %s -check-prefix=CUSTREAMATTACHMEMASYNC
// CUSTREAMATTACHMEMASYNC: CUDA API:
// CUSTREAMATTACHMEMASYNC-NEXT:   cuStreamAttachMemAsync(cs /*CUstream*/, d /*CUdeviceptr*/, s /*size_t*/,
// CUSTREAMATTACHMEMASYNC-NEXT:                          u /*unsigned int*/);
// CUSTREAMATTACHMEMASYNC-NEXT: The API is Removed.
// CUSTREAMATTACHMEMASYNC-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamCreate | FileCheck %s -check-prefix=CUSTREAMCREATE
// CUSTREAMCREATE: CUDA API:
// CUSTREAMCREATE-NEXT:   cuStreamCreate(ps /*CUstream **/, u /*unsigned int*/);
// CUSTREAMCREATE-NEXT: Is migrated to:
// CUSTREAMCREATE-NEXT:   *(ps) = dpct::get_current_device().create_queue();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamDestroy | FileCheck %s -check-prefix=CUSTREAMDESTROY
// CUSTREAMDESTROY: CUDA API:
// CUSTREAMDESTROY-NEXT:   cuStreamDestroy(s /*CUstream*/);
// CUSTREAMDESTROY-NEXT: Is migrated to:
// CUSTREAMDESTROY-NEXT:   dpct::get_current_device().destroy_queue(s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamQuery | FileCheck %s -check-prefix=CUSTREAMQUERY
// CUSTREAMQUERY: CUDA API:
// CUSTREAMQUERY-NEXT:   CUstream s;
// CUSTREAMQUERY-NEXT:   cuStreamQuery(s);
// CUSTREAMQUERY-NEXT: Is migrated to:
// CUSTREAMQUERY-NEXT:   dpct::queue_ptr s;
// CUSTREAMQUERY-NEXT:   s->ext_oneapi_empty();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamSynchronize | FileCheck %s -check-prefix=CUSTREAMSYNCHRONIZE
// CUSTREAMSYNCHRONIZE: CUDA API:
// CUSTREAMSYNCHRONIZE-NEXT:   CUstream s;
// CUSTREAMSYNCHRONIZE-NEXT:   cuStreamSynchronize(s /*CUstream*/);
// CUSTREAMSYNCHRONIZE-NEXT: Is migrated to:
// CUSTREAMSYNCHRONIZE-NEXT:   dpct::queue_ptr s;
// CUSTREAMSYNCHRONIZE-NEXT:   s->wait();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuStreamWaitEvent | FileCheck %s -check-prefix=CUSTREAMWAITEVENT
// CUSTREAMWAITEVENT: CUDA API:
// CUSTREAMWAITEVENT-NEXT:   CUstream s;
// CUSTREAMWAITEVENT-NEXT:   cuStreamWaitEvent(s /*CUstream*/, e /*CUevent*/, u /*unsigned int*/);
// CUSTREAMWAITEVENT-NEXT: Is migrated to:
// CUSTREAMWAITEVENT-NEXT:   dpct::queue_ptr s;
// CUSTREAMWAITEVENT-NEXT:   s->ext_oneapi_submit_barrier({*e});

/// Event Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventCreate | FileCheck %s -check-prefix=CUEVENTCREATE
// CUEVENTCREATE: CUDA API:
// CUEVENTCREATE-NEXT:   cuEventCreate(pe /*CUevent **/, u /*unsigned int*/);
// CUEVENTCREATE-NEXT: Is migrated to:
// CUEVENTCREATE-NEXT:   *pe = new sycl::event();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventDestroy | FileCheck %s -check-prefix=CUEVENTDESTROY
// CUEVENTDESTROY: CUDA API:
// CUEVENTDESTROY-NEXT:   cuEventDestroy(e /*CUevent*/);
// CUEVENTDESTROY-NEXT: Is migrated to:
// CUEVENTDESTROY-NEXT:   dpct::destroy_event(e);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventElapsedTime | FileCheck %s -check-prefix=CUEVENTELAPSEDTIME
// CUEVENTELAPSEDTIME: CUDA API:
// CUEVENTELAPSEDTIME-NEXT:   cuEventElapsedTime(pf /*float **/, e1 /*CUevent*/, e2 /*CUevent*/);
// CUEVENTELAPSEDTIME-NEXT: Is migrated to:
// CUEVENTELAPSEDTIME-NEXT:   *(pf) = std::chrono::duration<float, std::milli>(e2_ct1 - e1_ct1).count();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventQuery | FileCheck %s -check-prefix=CUEVENTQUERY
// CUEVENTQUERY: CUDA API:
// CUEVENTQUERY-NEXT:   CUevent e;
// CUEVENTQUERY-NEXT:   cuEventQuery(e /*CUevent*/);
// CUEVENTQUERY-NEXT: Is migrated to:
// CUEVENTQUERY-NEXT:   dpct::event_ptr e;
// CUEVENTQUERY-NEXT:   dpct::sycl_event_query(e /*CUevent*/);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventRecord | FileCheck %s -check-prefix=CUEVENTRECORD
// CUEVENTRECORD: CUDA API:
// CUEVENTRECORD-NEXT:   CUstream s;
// CUEVENTRECORD-NEXT:   cuEventRecord(e /*CUevent*/, s /*CUstream*/);
// CUEVENTRECORD-NEXT: Is migrated to:
// CUEVENTRECORD-NEXT:   dpct::queue_ptr s;
// CUEVENTRECORD-NEXT:   *e = s->ext_oneapi_submit_barrier();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuEventSynchronize | FileCheck %s -check-prefix=CUEVENTSYNCHRONIZE
// CUEVENTSYNCHRONIZE: CUDA API:
// CUEVENTSYNCHRONIZE-NEXT:   CUevent e;
// CUEVENTSYNCHRONIZE-NEXT:   cuEventSynchronize(e /*CUevent*/);
// CUEVENTSYNCHRONIZE-NEXT: Is migrated to:
// CUEVENTSYNCHRONIZE-NEXT:   dpct::event_ptr e;
// CUEVENTSYNCHRONIZE-NEXT:   e->wait_and_throw();

/// Execution Control

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuFuncGetAttribute | FileCheck %s -check-prefix=CUFUNCGETATTRIBUTE
// CUFUNCGETATTRIBUTE: CUDA API:
// CUFUNCGETATTRIBUTE-NEXT:   cuFuncGetAttribute(pi /*int **/, fa /*CUfunction_attribute*/,
// CUFUNCGETATTRIBUTE-NEXT:                      f /*CUfunction*/);
// CUFUNCGETATTRIBUTE-NEXT: Is migrated to:
// CUFUNCGETATTRIBUTE-NEXT:   *pi = dpct::get_kernel_function_info(f).fa;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuFuncSetCacheConfig | FileCheck %s -check-prefix=CUFUNCSETCACHECONFIG
// CUFUNCSETCACHECONFIG: CUDA API:
// CUFUNCSETCACHECONFIG-NEXT:   cuFuncSetCacheConfig(f /*CUfunction*/, fc /*CUfunc_cache*/);
// CUFUNCSETCACHECONFIG-NEXT: The API is Removed.
// CUFUNCSETCACHECONFIG-EMPTY:

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuLaunchKernel | FileCheck %s -check-prefix=CULAUNCHKERNEL
// CULAUNCHKERNEL: CUDA API:
// CULAUNCHKERNEL-NEXT:   cuLaunchKernel(f /*CUfunction*/, u1 /*unsigned int*/, u2 /*unsigned int*/,
// CULAUNCHKERNEL-NEXT:                  u3 /*unsigned int*/, u4 /*unsigned int*/, u5 /*unsigned int*/,
// CULAUNCHKERNEL-NEXT:                  u6 /*unsigned int*/, u7 /*unsigned int*/, s /*CUstream*/,
// CULAUNCHKERNEL-NEXT:                  pParam /*void ***/, pOpt /*void ***/);
// CULAUNCHKERNEL-NEXT: Is migrated to:
// CULAUNCHKERNEL-NEXT:   dpct::invoke_kernel_function(f, *s, sycl::range<3>(u3, u2, u1), sycl::range<3>(u6, u5, u4), u7, pParam, pOpt);

/// Occupancy

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuOccupancyMaxActiveBlocksPerMultiprocessor | FileCheck %s -check-prefix=CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR
// CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR: CUDA API:
// CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR-NEXT:   cuOccupancyMaxActiveBlocksPerMultiprocessor(pi /*int **/, f /*CUfunction*/,
// CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR-NEXT:                                               i /*int*/, s /*size_t*/);
// CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR-NEXT: Is migrated to (with the option --use-experimental-features=occupancy-calculation):
// CUOCCUPANCYMAXACTIVEBLOCKSPERMULTIPROCESSOR-NEXT:   dpct::experimental::calculate_max_active_wg_per_xecore(pi, i, s + dpct_placeholder /* total share local memory size */);

/// Texture Reference Management [DEPRECATED]

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetAddressMode | FileCheck %s -check-prefix=CUTEXREFGETADDRESSMODE
// CUTEXREFGETADDRESSMODE: CUDA API:
// CUTEXREFGETADDRESSMODE-NEXT:   CUtexref t;
// CUTEXREFGETADDRESSMODE-NEXT:   cuTexRefGetAddressMode(pa /*CUaddress_mode **/, t /*CUtexref*/, i /*int*/);
// CUTEXREFGETADDRESSMODE-NEXT: Is migrated to:
// CUTEXREFGETADDRESSMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETADDRESSMODE-NEXT:   *pa = t->get_addressing_mode();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetFilterMode | FileCheck %s -check-prefix=CUTEXREFGETFILTERMODE
// CUTEXREFGETFILTERMODE: CUDA API:
// CUTEXREFGETFILTERMODE-NEXT:   CUtexref t;
// CUTEXREFGETFILTERMODE-NEXT:   cuTexRefGetFilterMode(pf /*CUfilter_mode **/, t /*CUtexref*/);
// CUTEXREFGETFILTERMODE-NEXT: Is migrated to:
// CUTEXREFGETFILTERMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETFILTERMODE-NEXT:   *pf = t->get_filtering_mode();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefGetFlags | FileCheck %s -check-prefix=CUTEXREFGETFLAGS
// CUTEXREFGETFLAGS: CUDA API:
// CUTEXREFGETFLAGS-NEXT:   CUtexref t;
// CUTEXREFGETFLAGS-NEXT:   cuTexRefGetFlags(pu /*unsigned int **/, t /*CUtexref*/);
// CUTEXREFGETFLAGS-NEXT: Is migrated to:
// CUTEXREFGETFLAGS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFGETFLAGS-NEXT:   *pu = t->is_coordinate_normalized() << 1;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddress | FileCheck %s -check-prefix=CUTEXREFSETADDRESS
// CUTEXREFSETADDRESS: CUDA API:
// CUTEXREFSETADDRESS-NEXT:   CUtexref t;
// CUTEXREFSETADDRESS-NEXT:   cuTexRefSetAddress(ps /*size_t **/, t /*CUtexref*/, d /*CUdeviceptr*/,
// CUTEXREFSETADDRESS-NEXT:                      s /*size_t*/);
// CUTEXREFSETADDRESS-NEXT: Is migrated to:
// CUTEXREFSETADDRESS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESS-NEXT:   t->attach(d, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddress2D | FileCheck %s -check-prefix=CUTEXREFSETADDRESS2D
// CUTEXREFSETADDRESS2D: CUDA API:
// CUTEXREFSETADDRESS2D-NEXT:   CUtexref t;
// CUTEXREFSETADDRESS2D-NEXT:   cuTexRefSetAddress2D(t /*CUtexref*/, pa /*const CUDA_ARRAY_DESCRIPTOR **/,
// CUTEXREFSETADDRESS2D-NEXT:                        d /*CUdeviceptr*/, s /*size_t*/);
// CUTEXREFSETADDRESS2D-NEXT: Is migrated to:
// CUTEXREFSETADDRESS2D-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESS2D-NEXT:   t->attach(pa, d, s);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetAddressMode | FileCheck %s -check-prefix=CUTEXREFSETADDRESSMODE
// CUTEXREFSETADDRESSMODE: CUDA API:
// CUTEXREFSETADDRESSMODE-NEXT:   CUtexref t;
// CUTEXREFSETADDRESSMODE-NEXT:   cuTexRefSetAddressMode(t /*CUtexref*/, i /*int **/, a /*CUaddress_mode*/);
// CUTEXREFSETADDRESSMODE-NEXT: Is migrated to:
// CUTEXREFSETADDRESSMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETADDRESSMODE-NEXT:   t->set(a);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetArray | FileCheck %s -check-prefix=CUTEXREFSETARRAY
// CUTEXREFSETARRAY: CUDA API:
// CUTEXREFSETARRAY-NEXT:   CUtexref t;
// CUTEXREFSETARRAY-NEXT:   cuTexRefSetArray(t /*CUtexref*/, a /*CUarray*/, u /*unsigned int*/);
// CUTEXREFSETARRAY-NEXT: Is migrated to:
// CUTEXREFSETARRAY-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETARRAY-NEXT:   t->attach(dpct::image_data(a));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFilterMode | FileCheck %s -check-prefix=CUTEXREFSETFILTERMODE
// CUTEXREFSETFILTERMODE: CUDA API:
// CUTEXREFSETFILTERMODE-NEXT:   CUtexref t;
// CUTEXREFSETFILTERMODE-NEXT:   cuTexRefSetFilterMode(t /*CUtexref*/, f /*CUfilter_mode*/);
// CUTEXREFSETFILTERMODE-NEXT: Is migrated to:
// CUTEXREFSETFILTERMODE-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFILTERMODE-NEXT:   t->set(f);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFlags | FileCheck %s -check-prefix=CUTEXREFSETFLAGS
// CUTEXREFSETFLAGS: CUDA API:
// CUTEXREFSETFLAGS-NEXT:   CUtexref t;
// CUTEXREFSETFLAGS-NEXT:   cuTexRefSetFlags(t /*CUtexref*/, u /*unsigned int*/);
// CUTEXREFSETFLAGS-NEXT: Is migrated to:
// CUTEXREFSETFLAGS-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFLAGS-NEXT:   t->set_coordinate_normalization_mode(u & 0x02);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexRefSetFormat | FileCheck %s -check-prefix=CUTEXREFSETFORMAT
// CUTEXREFSETFORMAT: CUDA API:
// CUTEXREFSETFORMAT-NEXT:   CUtexref t;
// CUTEXREFSETFORMAT-NEXT:   cuTexRefSetFormat(t /*CUtexref*/, a /*CUarray_format*/, i /*int*/);
// CUTEXREFSETFORMAT-NEXT: Is migrated to:
// CUTEXREFSETFORMAT-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXREFSETFORMAT-NEXT:   t->set_channel_type(a);
// CUTEXREFSETFORMAT-NEXT:   t->set_channel_num(i);

/// Texture Object Management

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectCreate | FileCheck %s -check-prefix=CUTEXOBJECTCREATE
// CUTEXOBJECTCREATE: CUDA API:
// CUTEXOBJECTCREATE-NEXT:   cuTexObjectCreate(pt /*CUtexObject **/, pr /*const CUDA_RESOURCE_DESC **/,
// CUTEXOBJECTCREATE-NEXT:                     ptd /*const CUDA_TEXTURE_DESC **/,
// CUTEXOBJECTCREATE-NEXT:                     prv /*const CUDA_RESOURCE_VIEW_DESC **/);
// CUTEXOBJECTCREATE-NEXT: Is migrated to:
// CUTEXOBJECTCREATE-NEXT:   *pt = dpct::create_image_wrapper(*pr, *ptd);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectDestroy | FileCheck %s -check-prefix=CUTEXOBJECTDESTROY
// CUTEXOBJECTDESTROY: CUDA API:
// CUTEXOBJECTDESTROY-NEXT:   cuTexObjectDestroy(t /*CUtexObject*/);
// CUTEXOBJECTDESTROY-NEXT: Is migrated to:
// CUTEXOBJECTDESTROY-NEXT:   delete t;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectGetResourceDesc | FileCheck %s -check-prefix=CUTEXOBJECTGETRESOURCEDESC
// CUTEXOBJECTGETRESOURCEDESC: CUDA API:
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   CUtexObject t;
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   cuTexObjectGetResourceDesc(pr /*CUDA_RESOURCE_DESC **/, t /*CUtexObject*/);
// CUTEXOBJECTGETRESOURCEDESC-NEXT: Is migrated to:
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXOBJECTGETRESOURCEDESC-NEXT:   *pr = t->get_data();

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuTexObjectGetTextureDesc | FileCheck %s -check-prefix=CUTEXOBJECTGETTEXTUREDESC
// CUTEXOBJECTGETTEXTUREDESC: CUDA API:
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   CUtexObject t;
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   cuTexObjectGetTextureDesc(pt /*CUDA_TEXTURE_DESC **/, t /*CUtexObject*/);
// CUTEXOBJECTGETTEXTUREDESC-NEXT: Is migrated to:
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   dpct::image_wrapper_base_p t;
// CUTEXOBJECTGETTEXTUREDESC-NEXT:   *pt = t->get_sampling_info();

/// Peer Context Memory Access

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuCtxEnablePeerAccess | FileCheck %s -check-prefix=CUCTXENABLEPEERACCESS
// CUCTXENABLEPEERACCESS: CUDA API:
// CUCTXENABLEPEERACCESS-NEXT:   cuCtxEnablePeerAccess(c /*CUcontext*/, u /*unsigned*/);
// CUCTXENABLEPEERACCESS-NEXT: Is migrated to:
// CUCTXENABLEPEERACCESS-NEXT:   dpct::get_current_device().ext_oneapi_enable_peer_access(dpct::get_device(c));

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cuDeviceCanAccessPeer | FileCheck %s -check-prefix=CUDEVICECANACCESSPEER
// CUDEVICECANACCESSPEER: CUDA API:
// CUDEVICECANACCESSPEER-NEXT:   cuDeviceCanAccessPeer(pi /*int **/, d1 /*CUdevice*/, d2 /*CUdevice*/);
// CUDEVICECANACCESSPEER-NEXT: Is migrated to:
// CUDEVICECANACCESSPEER-NEXT:   *pi = dpct::get_device(d1).ext_oneapi_can_access_peer(dpct::get_device(d2));
