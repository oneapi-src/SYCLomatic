
#include <hip/hip_runtime.h>
// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cudaStreamGetFlags | FileCheck %s -check-prefix=CUDASTREAMGETFLAGS
// CUDASTREAMGETFLAGS: CUDA API: cudaStreamGetFlags(s /*cudaStream_t*/, f /*unsigned int **/);
// CUDASTREAMGETFLAGS-NEXT: Is migrated to: *(f) = 0;

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=cudaEventDestroy | FileCheck %s -check-prefix=CUDAEVENTDESTROY
// CUDAEVENTDESTROY: CUDA API: cudaEventDestroy(e /*cudaEvent_t*/);
// CUDAEVENTDESTROY-NEXT: Is migrated to: dpct::destroy_event(e);

// RUN: dpct --cuda-include-path="%cuda-path/include" --query-api-mapping=aaa | FileCheck %s -check-prefix=AAA
// AAA: The API Mapping is not available
