// RUN: dpct --format-range=none -out-root %T/array_memory_management %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/array_memory_management/array_memory_management.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/array_memory_management/array_memory_management.dp.cpp -o %T/array_memory_management/array_memory_management.dp.o %}

#include <hip/hip_runtime.h>

#define CHECK_ERR(x) do {                         \
  hipError_t err = x;                            \
  if (err != hipSuccess) {                          \
    return; \
  }                                                  \
} while(0)

void checkError(hipError_t err) {
}

class C {
public:
  int *data{nullptr};
};

void foo() {
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  C c;
  // CHECK: dpct::queue_ptr s;
  // CHECK-NEXT: dpct::image_matrix_p a1;
  // CHECK-NEXT: dpct::image_matrix* a2;
  // CHECK-NEXT: dpct::err0 err;
  // CHECK-NEXT: sycl::range<3> extent{0, 0, 0};
  // CHECK-NEXT: dpct::image_channel channel;
  hipStream_t s;
  hipArray_t a1;
  hipArray* a2;
  hipError_t err;
  hipExtent extent;
  hipChannelFormatDesc channel;

  // CHECK: a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height));
  hipMallocArray(&a1, &channel, width, height);

  // CHECK: a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height));
  hipMallocArray(&a1, &channel, width, height, 0);

  // CHECK: a1 = new dpct::image_matrix(channel, extent);
  hipMalloc3DArray(&a1, &channel, extent);

  // CHECK: a1 = new dpct::image_matrix(channel, extent);
  hipMalloc3DArray(&a1, &channel, extent, 0);

  // CHECK: dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height);

  // CHECK: dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1), dpct::automatic, *s);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(((dpct::image_matrix *)c.data)->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyToArray((hipArray *)c.data, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1), dpct::automatic, *s);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width);

  // CHECK: delete a1;
  hipFreeArray(a1);
  // CHECK: delete a2;
  hipFreeArray(a2);

  // CHECK:  err = DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height)));
  err = hipMallocArray(&a1, &channel, width, height);
  // CHECK:  checkError(DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height))));
  checkError(hipMallocArray(&a1, &channel, width, height));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height))));
  CHECK_ERR(hipMallocArray(&a1, &channel, width, height));

  // CHECK:  err = DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, extent));
  err = hipMalloc3DArray(&a1, &channel, extent);
  // CHECK:  checkError(DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, extent)));
  checkError(hipMalloc3DArray(&a1, &channel, extent));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(a1 = new dpct::image_matrix(channel, extent)));
  CHECK_ERR(hipMalloc3DArray(&a1, &channel, extent));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)));
  err = hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  checkError(hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  CHECK_ERR(hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s));
  err = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s)));
  checkError(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s)));
  CHECK_ERR(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)));
  err = hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  checkError(hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  CHECK_ERR(hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)));
  err = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  checkError(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1))));
  CHECK_ERR(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)));
  err = hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  checkError(hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1))));
  CHECK_ERR(hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  err = hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  checkError(hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  CHECK_ERR(hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  err = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  checkError(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  CHECK_ERR(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)));
  err = hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  checkError(hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  CHECK_ERR(hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)));
  err = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  checkError(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1))));
  CHECK_ERR(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));

  // CHECK:  err = DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)));
  err = cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:  checkError(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  checkError(cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1))));
  CHECK_ERR(cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost));

  hipStream_t cs;
  hipArray_t acu;
  // CHECK: dpct::dpct_memcpy((char *)(acu->to_pitched_data().get_data_ptr()) + woffset, data, width);
  hipMemcpyHtoA(acu, woffset, data, width);
  // CHECK: dpct::dpct_memcpy(data, (char *)(acu->to_pitched_data().get_data_ptr()) + woffset, width);
  hipMemcpyAtoH(data, acu, woffset, width);
  // CHECK: dpct::async_dpct_memcpy((char *)(acu->to_pitched_data().get_data_ptr()) + woffset, data, width, dpct::automatic, *cs);
  hipMemcpyHtoAAsync(acu, woffset, data, width, cs);
  // CHECK: dpct::async_dpct_memcpy(data, (char *)(acu->to_pitched_data().get_data_ptr()) + woffset, width, dpct::automatic, *cs);
  hipMemcpyAtoHAsync(data, acu, woffset, width, cs);

  hipDeviceptr_t data2;
  hipMalloc(&data2, sizeof(int) * 30);
  // CHECK: dpct::dpct_memcpy((char *)(acu->to_pitched_data().get_data_ptr()) + woffset, data2, width);
  hipMemcpyDtoA(acu, woffset, data2, width);
  // CHECK: dpct::dpct_memcpy(data2, (char *)(acu->to_pitched_data().get_data_ptr()) + woffset, width);
  hipMemcpyAtoD(data2, acu, woffset, width);

  hipArray_t acu2;
  // CHECK: dpct::dpct_memcpy((char *)(acu->to_pitched_data().get_data_ptr()) + woffset, (char *)(acu2->to_pitched_data().get_data_ptr()) + woffset, width);
  hipMemcpyAtoA(acu, woffset, acu2, woffset, width);

  // CHECK:  err = DPCT_CHECK_ERROR(delete a1);
  err = hipFreeArray(a1);
  // CHECK:  checkError(DPCT_CHECK_ERROR(delete a1));
  checkError(hipFreeArray(a1));
  // CHECK:  CHECK_ERR(DPCT_CHECK_ERROR(delete a1));
  CHECK_ERR(hipFreeArray(a1));
}

void foo2() {
  // CHECK: dpct::memcpy_parameter p1 = {};
  // CHECK-NEXT: dpct::image_matrix_p *a1;
  // CHECK-NEXT: p1.to.image = *a1;
  hipMemcpy3DParms p1 = {0};
  hipArray_t *a1;
  p1.dstArray = *a1;

  // CHECK: dpct::memcpy_parameter p2 = {};
  // CHECK-NEXT: dpct::image_matrix_p *a2;
  // CHECK-NEXT: p2.to.image = *a2;
  HIP_MEMCPY3D p2 = {0};
  hipArray_t *a2;
  p2.dstArray = *a2;

  // CHECK: dpct::memcpy_parameter p3 = {};
  // CHECK-NEXT: dpct::image_matrix **a3;
  // CHECK-NEXT: p3.to.image = *a3;
  HIP_MEMCPY3D p3 = {0};
  hipArray **a3;
  p3.dstArray = *a3;
}
