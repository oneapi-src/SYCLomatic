// RUN: dpct --format-range=none -out-root %T/array_memory_management %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck --input-file %T/array_memory_management/array_memory_management.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>

#define CHECK_ERR(x) do {                         \
  hipError_t err = x;                            \
  if (err != hipSuccess) {                          \
    return; \
  }                                                  \
} while(0)

void checkError(hipError_t err) {
}

class C {
public:
  int *data{nullptr};
};

void foo() {
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  C c;
  // CHECK: dpct::queue_ptr s;
  // CHECK-NEXT: dpct::image_matrix_p a1;
  // CHECK-NEXT: dpct::image_matrix* a2;
  // CHECK-NEXT: dpct::err0 err;
  // CHECK-NEXT: sycl::range<3> extent{0, 0, 0};
  // CHECK-NEXT: dpct::image_channel channel;
  hipStream_t s;
  hipArray_t a1;
  hipArray* a2;
  hipError_t err;
  hipExtent extent;
  hipChannelFormatDesc channel;

  // CHECK: a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height));
  hipMallocArray(&a1, &channel, width, height);

  // CHECK: a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height));
  hipMallocArray(&a1, &channel, width, height, 0);

  // CHECK: a1 = new dpct::image_matrix(channel, extent);
  hipMalloc3DArray(&a1, &channel, extent);

  // CHECK: a1 = new dpct::image_matrix(channel, extent);
  hipMalloc3DArray(&a1, &channel, extent, 0);

  // CHECK: dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height);

  // CHECK: dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1), dpct::automatic, *s);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(((dpct::image_matrix *)c.data)->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  hipMemcpyToArray((hipArray *)c.data, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, 0);

  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1), dpct::automatic, *s);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, s);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost);

  // CHECK: dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width);

  // CHECK: delete a1;
  hipFreeArray(a1);
  // CHECK: delete a2;
  hipFreeArray(a2);

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height)), 0);
  err = hipMallocArray(&a1, &channel, width, height);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height)), 0));
  checkError(hipMallocArray(&a1, &channel, width, height));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((a1 = new dpct::image_matrix(channel, sycl::range<2>(width, height)), 0));
  CHECK_ERR(hipMallocArray(&a1, &channel, width, height));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (a1 = new dpct::image_matrix(channel, extent), 0);
  err = hipMalloc3DArray(&a1, &channel, extent);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((a1 = new dpct::image_matrix(channel, extent), 0));
  checkError(hipMalloc3DArray(&a1, &channel, extent));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((a1 = new dpct::image_matrix(channel, extent), 0));
  CHECK_ERR(hipMalloc3DArray(&a1, &channel, extent));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  err = hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  checkError(hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  CHECK_ERR(hipMemcpy2DFromArray(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s), 0);
  err = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s), 0));
  checkError(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1), dpct::automatic, *s), 0));
  CHECK_ERR(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, s));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  err = hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  checkError(hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  CHECK_ERR(hipMemcpy2DToArray(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  err = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  checkError(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  CHECK_ERR(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, 0));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  err = hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  checkError(hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  CHECK_ERR(hipMemcpy2DArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, height));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  err = hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  checkError(hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  CHECK_ERR(hipMemcpyFromArray(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  err = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  checkError(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  CHECK_ERR(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  err = hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  checkError(hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  CHECK_ERR(hipMemcpyToArray(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  err = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  checkError(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  CHECK_ERR(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  err = cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  checkError(cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((dpct::dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), a2->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  CHECK_ERR(cudaMemcpyArrayToArray(a1, woffset, hoffset, a2, woffset, hoffset, width, hipMemcpyDeviceToHost));

  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  err = (delete a1, 0);
  err = hipFreeArray(a1);
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  checkError((delete a1, 0));
  checkError(hipFreeArray(a1));
  // CHECK:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:  CHECK_ERR((delete a1, 0));
  CHECK_ERR(hipFreeArray(a1));
}

