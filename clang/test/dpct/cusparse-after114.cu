// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3
// RUN: dpct --format-range=none --out-root %T/cusparse-after114 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cusparse-after114/cusparse-after114.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusparse-after114/cusparse-after114.dp.cpp -o %T/cusparse-after114/cusparse-after114.dp.o %}


#include <hip/hip_runtime.h>
#include <hipsparse.h>

void foo1() {
  //CHECK:int spsmDescr;
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpSM_createDescr was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT:*/
  hipsparseSpSMDescr_t spsmDescr;
  hipsparseSpSM_createDescr(&spsmDescr);

  hipsparseHandle_t handle;
  hipsparseOperation_t opA;
  hipsparseOperation_t opB;
  const void *alpha;
  //CHECK:dpct::sparse::sparse_matrix_desc_t matA;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::dense_matrix_desc> matB;
  hipsparseConstSpMatDescr_t matA;
  hipsparseConstDnMatDescr_t matB;
  hipsparseDnMatDescr_t matC;
  hipDataType computeType;
  //CHECK:int alg;
  hipsparseSpSMAlg_t alg;
  size_t bufferSize;
  void *externalBuffer;

  hipsparseStatus_t status;
  //CHECK:status = DPCT_CHECK_ERROR(bufferSize = 0);
  //CHECK-NEXT:status = DPCT_CHECK_ERROR(dpct::sparse::spsm_optimize(handle->get_queue(), opA, matA, matB, matC));
  //CHECK-NEXT:status = DPCT_CHECK_ERROR(dpct::sparse::spsm(handle->get_queue(), opA, opB, alpha, matA, matB, matC, computeType));
  status = hipsparseSpSM_bufferSize(handle, opA, opB, alpha, matA, matB, matC, computeType, alg, spsmDescr, &bufferSize);
  status = hipsparseSpSM_analysis(handle, opA, opB, alpha, matA, matB, matC, computeType, alg, spsmDescr, externalBuffer);
  status = hipsparseSpSM_solve(handle, opA, opB, alpha, matA, matB, matC, computeType, alg, spsmDescr, nullptr);

  //CHECK:/*
  //CHECK-NEXT:DPCT1026:{{[0-9]+}}: The call to cusparseSpSM_destroyDescr was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT:*/
  hipsparseSpSM_destroyDescr(spsmDescr);
}
