// RUN: dpct --format-range=none -usm-level=none -out-root %T/cudamallocmanaged_none_usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++14
// RUN: FileCheck %s --match-full-lines --input-file %T/cudamallocmanaged_none_usm/cudamallocmanaged_none_usm.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/cudamallocmanaged_none_usm/cudamallocmanaged_none_usm.dp.cpp -o %T/cudamallocmanaged_none_usm/cudamallocmanaged_none_usm.dp.o %}

#define VECTOR_SIZE 256
#include<hip/hip_runtime.h>
#include<stdio.h>

__global__ void ker(int *A, int *B, int *C){
  C[threadIdx.x] = A[threadIdx.x] + B[threadIdx.x];
}

struct A{
  struct B{
    struct C{
      int *a;
    };
    C c;
  };
  B b;
};

template<typename T>
void temp1(){
  T* a;
  T b = 0;

  // CHECK: a = (T *)dpct::dpct_malloc(10 * sizeof(T));
  hipMallocManaged(&a, 10 * sizeof(T));

  // CHECK: b = dpct::get_host_ptr<T>(a)[2];
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(a++);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(++a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(a--);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(--a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(a + 2);
  // CHECK-NEXT: b = *dpct::get_host_ptr<T>(a) + 1;
  b = a[2];
  b = *a;
  b = *a++;
  b = *++a;
  b = *a--;
  b = *--a;
  b = *(a + 2);
  b = *a + 1;

  // CHECK: dpct::get_host_ptr<T>(a)[2] = b;
  // CHECK-NEXT: *dpct::get_host_ptr<T>(a) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<T>(a++) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<T>(++a) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<T>(a--) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<T>(a + 2) = b;
  a[2] = b;
  *a = b;
  *a++ = b;
  *++a = b;
  *a-- = b;
  *(a + 2) = b;

  hipFree(a);
}

template<typename T>
void temp2(){
  T* a, *b, *c;
  // CHECK: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<T>(a) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (T *)dpct::dpct_malloc(10 * sizeof(T));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'b' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<T>(b) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: b = (T *)dpct::dpct_malloc(10 * sizeof(T));
  // CHECK-NEXT: a = (T *)dpct::dpct_malloc(10 * sizeof(T));
  hipMallocManaged(&a, 10 * sizeof(T));
  hipMallocManaged(&b, 10 * sizeof(T));
  hipMalloc(&a, 10 * sizeof(T));

  c = b;
  // CHECK: a[0] = 1;
  a[0] = 1;
  // CHECK: dpct::get_host_ptr<T>(b)[0] = 1;
  b[0] = 1;

  hipFree(a);
  hipFree(b);
}

template<typename T>
void temp3(){
  A aa;
  // CHECK: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'aa.b.c.a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<int>(aa.b.c.a) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: aa.b.c.a = (int *)dpct::dpct_malloc(10 * sizeof(int));
  hipMallocManaged(&aa.b.c.a, 10 * sizeof(int));

  // CHECK: aa.b.c.a[0] = 1;
  aa.b.c.a[0] = 1;

  hipFree(aa.b.c.a);
}

class A1{
  int* a;
public:
  A1(){
    // CHECK: a = (int *)dpct::dpct_malloc(10 * sizeof(int));
    hipMallocManaged(&a, 10 * sizeof(int));
  };
  void run1(){
    int b;
    // CHECK: b = dpct::get_host_ptr<int>(a)[2];
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a++);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(++a);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a--);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(--a);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a + 2);
    // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a) + 1;
    b = a[2];
    b = *a;
    b = *a++;
    b = *++a;
    b = *a--;
    b = *--a;
    b = *(a + 2);
    b = *a + 1;

    // CHECK: dpct::get_host_ptr<int>(a)[2] = b;
    // CHECK-NEXT: *dpct::get_host_ptr<int>(a) = b;
    // CHECK-NEXT: *dpct::get_host_ptr<int>(a++) = b;
    // CHECK-NEXT: *dpct::get_host_ptr<int>(++a) = b;
    // CHECK-NEXT: *dpct::get_host_ptr<int>(a--) = b;
    // CHECK-NEXT: *dpct::get_host_ptr<int>(a + 2) = b;
    a[2] = b;
    *a = b;
    *a++ = b;
    *++a = b;
    *a-- = b;
    *(a + 2) = b;
  };
  ~A1(){
    hipFree(a);
  }
};

class A2{
  int* a;
  int* b;
  int* c;
public:
  A2(){
    // CHECK: /*
    // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<int>(a) to access the pointer from the host code.
    // CHECK-NEXT: */
    // CHECK-NEXT: a = (int *)dpct::dpct_malloc(10 * sizeof(int));
    // CHECK-NEXT: /*
    // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'b' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<int>(b) to access the pointer from the host code.
    // CHECK-NEXT: */
    // CHECK-NEXT: b = (int *)dpct::dpct_malloc(10 * sizeof(int));
    hipMallocManaged(&a, 10 * sizeof(int));
    hipMallocManaged(&b, 10 * sizeof(int));

  };
  void run2(){

    hipMalloc(&a, 10 * sizeof(int));

    c = b;
    // CHECK: a[0] = 1;
    a[0] = 1;
    // CHECK: dpct::get_host_ptr<int>(b)[0] = 1;
    b[0] = 1;

  };
  ~A2(){
    hipFree(a);
    hipFree(b);
  }
};

class A3{
  A aa;
public:
  A3(){
    A aa;
    // CHECK: /*
    // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'aa.b.c.a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<int>(aa.b.c.a) to access the pointer from the host code.
    // CHECK-NEXT: */
    // CHECK-NEXT: aa.b.c.a = (int *)dpct::dpct_malloc(10 * sizeof(int));
    hipMallocManaged(&aa.b.c.a, 10 * sizeof(int));
  }
  void run3(){
    // CHECK: aa.b.c.a[0] = 1;
    aa.b.c.a[0] = 1;
  };
  ~A3(){
    hipFree(aa.b.c.a);
  }
};

void test1(){
  int *a, b;
  // CHECK: a = (int *)dpct::dpct_malloc(10 * sizeof(int));
  hipMallocManaged(&a, 10 * sizeof(int));

  // CHECK: b = dpct::get_host_ptr<int>(a)[2];
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a++);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(++a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a--);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(--a);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a + 2);
  // CHECK-NEXT: b = *dpct::get_host_ptr<int>(a) + 1;
  b = a[2];
  b = *a;
  b = *a++;
  b = *++a;
  b = *a--;
  b = *--a;
  b = *(a + 2);
  b = *a + 1;

  // CHECK: dpct::get_host_ptr<int>(a)[2] = b;
  // CHECK-NEXT: *dpct::get_host_ptr<int>(a) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<int>(a++) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<int>(++a) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<int>(a--) = b;
  // CHECK-NEXT: *dpct::get_host_ptr<int>(a + 2) = b;
  a[2] = b;
  *a = b;
  *a++ = b;
  *++a = b;
  *a-- = b;
  *(a + 2) = b;

  hipFree(a);
}

void test2(){
  float* a, *b, *c;
  // CHECK: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<float>(a) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (float *)dpct::dpct_malloc(10 * sizeof(float));
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'b' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<float>(b) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: b = (float *)dpct::dpct_malloc(10 * sizeof(float));
  hipMallocManaged(&a, 10 * sizeof(float));
  hipMallocManaged(&b, 10 * sizeof(float));

  hipMalloc(&a, 10 * sizeof(float));

  c = b;
  // CHECK: a[0] = 1;
  a[0] = 1;
  // CHECK: dpct::get_host_ptr<float>(b)[0] = 1;
  b[0] = 1;

  hipFree(a);
  hipFree(b);
}

void test3(){
  A aa;
  // CHECK: /*
  // CHECK-NEXT: DPCT1070:{{[0-9]+}}: 'aa.b.c.a' is allocated by dpct::dpct_malloc. Use dpct::get_host_ptr<int>(aa.b.c.a) to access the pointer from the host code.
  // CHECK-NEXT: */
  // CHECK-NEXT: aa.b.c.a = (int *)dpct::dpct_malloc(10 * sizeof(int));
  hipMallocManaged(&aa.b.c.a, 10 * sizeof(int));

  // CHECK: aa.b.c.a[0] = 1;
  aa.b.c.a[0] = 1;

  hipFree(aa.b.c.a);
}

int main(){
  int *a, *b, *c;

  // CHECK: a = (int *)dpct::dpct_malloc(VECTOR_SIZE * sizeof(float));
  // CHECK-NEXT: b = (int *)dpct::dpct_malloc(VECTOR_SIZE * sizeof(float));
  // CHECK-NEXT: c = (int *)dpct::dpct_malloc(VECTOR_SIZE * sizeof(float));
  hipMallocManaged(&a, VECTOR_SIZE * sizeof(float));
  hipMallocManaged(&b, VECTOR_SIZE * sizeof(float));
  hipMallocManaged(&c, VECTOR_SIZE * sizeof(float));

  for(int i = 0; i < VECTOR_SIZE; i++){
    // CHECK: dpct::get_host_ptr<int>(a)[i] = i;
    // CHECK-NEXT: dpct::get_host_ptr<int>(b)[i] = i;
    a[i] = i;
    b[i] = i;
  }

  ker<<<1, VECTOR_SIZE>>>(a, b, c);

  for(int i = 0; i < VECTOR_SIZE; i++){
    // CHECK: printf("%d", dpct::get_host_ptr<int>(c)[i]);
    printf("%d", c[i]);
  }
  hipFree(a);
  hipFree(b);
  hipFree(c);

  temp1<float>();
  temp2<int>();
  temp3<double>();
  test1();
  test2();
  test3();
  return 0;
}
