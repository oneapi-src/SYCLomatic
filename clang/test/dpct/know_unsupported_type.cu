// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.2
// RUN: dpct -out-root %T/know_unsupported_type %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/know_unsupported_type/know_unsupported_type.dp.cpp
#include "hip/hip_runtime.h"

#include "hipsparse.h"
#include "nvml.h"
#include <vector>
int main(int argc, char **argv) {
    // CHECK: dpct::matrix_desc *pcad;
    HIP_ARRAY_DESCRIPTOR *pcad;
    // CHECK: dpct::memcpy_parameter *p1c3d;
    hipMemcpy3DParms *p1c3d;
    // CHECK: const dpct::memcpy_parameter *p2c3d;
    const hipMemcpy3DParms *p2c3d;
    // CHECK: static dpct::memcpy_parameter *p3c3d;
    static hipMemcpy3DParms *p3c3d;
    // CHECK: static volatile dpct::memcpy_parameter *p4c3d;
    static volatile hipMemcpy3DParms *p4c3d;
    // CHECK: std::vector<dpct::memcpy_parameter *> vc3dp;
    std::vector<hipMemcpy3DParms *> vc3dp;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUexternalMemory type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUexternalMemory cum;
    hipExternalMemory_t cum;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUexternalSemaphore type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUexternalSemaphore cus;
    hipExternalSemaphore_t cus;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUgraph type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUgraph cug;
    hipGraph_t cug;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUgraphExec type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUgraphExec cuge;
    hipGraphExec_t cuge;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUgraphNode type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUgraphNode cugn;
    hipGraphNode_t cugn;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of CUgraphicsResource type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: CUgraphicsResource cugr;
    hipGraphicsResource_t cugr;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of nvmlDevice_t type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: nvmlDevice_t nvmld;
    nvmlDevice_t nvmld;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of nvmlReturn_t type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: nvmlReturn_t nvmlr;
    nvmlReturn_t nvmlr;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of nvmlMemory_t type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: nvmlMemory_t nvmlm;
    nvmlMemory_t nvmlm;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of nvmlValueType_t type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: nvmlValueType_t nvmlvt;
    nvmlValueType_t nvmlvt;
    // CHECK: /*
    // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of nvmlValue_t type is not supported.
    // CHECK-NEXT: */
    // CHECK-NEXT: nvmlValue_t nvmlv;
    nvmlValue_t nvmlv;

    return 0;
}

