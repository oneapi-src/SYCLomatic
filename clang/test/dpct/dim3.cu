// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --format-range=none -out-root %T/dim3 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/dim3/dim3.dp.cpp --match-full-lines %s

#include <hip/hip_runtime.h>

int main() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam0 = {};
  hipKernelNodeParams kernelNodeParam0 = {};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam1 = {0};
  hipKernelNodeParams kernelNodeParam1 = {0};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam2 = {0, {1, 1, 0}};
  hipKernelNodeParams kernelNodeParam2 = {0, 0};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam3 = {0, {1, 1, 0}, {1, 1, 0}};
  hipKernelNodeParams kernelNodeParam3 = {0, 0, 0};

  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam4{};
  hipKernelNodeParams kernelNodeParam4{};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam5{0};
  hipKernelNodeParams kernelNodeParam5{0};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam6{0, {1, 1, 0}};
  hipKernelNodeParams kernelNodeParam6{0, 0};
  // CHECK: /*
  // CHECK-NEXT: DPCT1082:{{[0-9]+}}: Migration of cudaKernelNodeParams type is not supported.
  // CHECK-NEXT: */
  // CHECK-NEXT: cudaKernelNodeParams kernelNodeParam7{0, {1, 1, 0}, {1, 1, 0}};
  hipKernelNodeParams kernelNodeParam7{0, 0, 0};
}
