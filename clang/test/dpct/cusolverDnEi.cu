// RUN: dpct --format-range=none --usm-level=none -out-root %T/cusolverDnEi %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusolverDnEi/cusolverDnEi.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusolverDnEi/cusolverDnEi.dp.cpp -o %T/cusolverDnEi/cusolverDnEi.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hipsolver.h>
#include <hip/hip_runtime.h>


int main(int argc, char *argv[])
{
    hipsolverHandle_t* cusolverH = NULL;
    hipsolverStatus_t status = HIPSOLVER_STATUS_SUCCESS;
    status = HIPSOLVER_STATUS_NOT_INITIALIZED;
    hipblasFillMode_t uplo = HIPBLAS_FILL_MODE_LOWER;
    hipblasOperation_t trans = HIPBLAS_OP_N;
    hipblasSideMode_t side = HIPBLAS_SIDE_LEFT;
    hipsolverEigMode_t jobz;

    int m = 0;
    int n = 0;
    int k = 0;
    int nrhs = 0;
    float A_f = 0;
    double A_d = 0.0;
    hipComplex A_c = make_hipComplex(1,0);
    hipDoubleComplex A_z = make_hipDoubleComplex(1,0);

    float B_f = 0;
    double B_d = 0.0;
    hipComplex B_c = make_hipComplex(1,0);
    hipDoubleComplex B_z = make_hipDoubleComplex(1,0);

    float D_f = 0;
    double D_d = 0.0;
    hipComplex D_c = make_hipComplex(1,0);
    hipDoubleComplex D_z = make_hipDoubleComplex(1,0);

    float E_f = 0;
    double E_d = 0.0;
    hipComplex E_c = make_hipComplex(1,0);
    hipDoubleComplex E_z = make_hipDoubleComplex(1,0);

    float TAU_f = 0;
    double TAU_d = 0.0;
    hipComplex TAU_c = make_hipComplex(1,0);
    hipDoubleComplex TAU_z = make_hipDoubleComplex(1,0);

    float TAUQ_f = 0;
    double TAUQ_d = 0.0;
    hipComplex TAUQ_c = make_hipComplex(1,0);
    hipDoubleComplex TAUQ_z = make_hipDoubleComplex(1,0);

    float TAUP_f = 0;
    double TAUP_d = 0.0;
    hipComplex TAUP_c = make_hipComplex(1,0);
    hipDoubleComplex TAUP_z = make_hipDoubleComplex(1,0);

    const float C_f = 0;
    const double C_d = 0.0;
    const hipComplex C_c = make_hipComplex(1,0);
    const hipDoubleComplex C_z = make_hipDoubleComplex(1,0);

    int lda = 0;
    int ldb = 0;
    const int ldc = 0;
    float workspace_f = 0;
    double workspace_d = 0;
    hipComplex workspace_c = make_hipComplex(1,0);
    hipDoubleComplex workspace_z = make_hipDoubleComplex(1,0);
    int Lwork = 0;
    int devInfo = 0;
    int devIpiv = 0;

    signed char jobu;
    signed char jobvt;

    float S_f = 0;
    double S_d = 0.0;
    hipComplex S_c = make_hipComplex(1,0);
    hipDoubleComplex S_z = make_hipDoubleComplex(1,0);

    float U_f = 0;
    double U_d = 0.0;
    hipComplex U_c = make_hipComplex(1,0);
    hipDoubleComplex U_z = make_hipDoubleComplex(1,0);
    int ldu;

    float VT_f = 0;
    double VT_d = 0.0;
    hipComplex VT_c = make_hipComplex(1,0);
    hipDoubleComplex VT_z = make_hipDoubleComplex(1,0);
    int ldvt;

    float Rwork_f = 0;
    double Rwork_d = 0.0;
    hipComplex Rwork_c = make_hipComplex(1,0);
    hipDoubleComplex Rwork_z = make_hipDoubleComplex(1,0);

    float W_f = 0;
    double W_d = 0.0;
    hipComplex W_c = make_hipComplex(1,0);
    hipDoubleComplex W_z = make_hipDoubleComplex(1,0);

    //CHECK: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<float>(**cusolverH, m, n, lda_ct));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<float>(**cusolverH, m, n, lda_ct);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAUQ_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAUQ_f);
    //CHECK-NEXT: auto TAUP_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAUP_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAUQ_f_buf_ct{{[0-9]+}}, TAUP_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAUQ_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAUQ_f);
    //CHECK-NEXT: auto TAUP_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAUP_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_f_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAUQ_f_buf_ct{{[0-9]+}}, TAUP_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSgebrd(*cusolverH, m, n, &A_f, lda, &D_f, &E_f, &TAUQ_f, &TAUP_f, &workspace_f, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<double>(**cusolverH, m, n, lda_ct));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<double>(**cusolverH, m, n, lda_ct);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAUQ_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAUQ_d);
    //CHECK-NEXT: auto TAUP_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAUP_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAUQ_d_buf_ct{{[0-9]+}}, TAUP_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAUQ_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAUQ_d);
    //CHECK-NEXT: auto TAUP_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAUP_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_d_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAUQ_d_buf_ct{{[0-9]+}}, TAUP_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDgebrd(*cusolverH, m, n, &A_d, lda, &D_d, &E_d, &TAUQ_d, &TAUP_d, &workspace_d, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<std::complex<float>>(**cusolverH, m, n, lda_ct));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<std::complex<float>>(**cusolverH, m, n, lda_ct);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAUQ_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAUQ_c);
    //CHECK-NEXT: auto TAUP_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAUP_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAUQ_c_buf_ct{{[0-9]+}}, TAUP_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAUQ_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAUQ_c);
    //CHECK-NEXT: auto TAUP_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAUP_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_c_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAUQ_c_buf_ct{{[0-9]+}}, TAUP_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCgebrd(*cusolverH, m, n, &A_c, lda, &D_f, &E_f, &TAUQ_c, &TAUP_c, &workspace_c, Lwork, &devInfo);

    //CHECK: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<std::complex<double>>(**cusolverH, m, n, lda_ct));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: std::int64_t lda_ct;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gebrd_scratchpad_size<std::complex<double>>(**cusolverH, m, n, lda_ct);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAUQ_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAUQ_z);
    //CHECK-NEXT: auto TAUP_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAUP_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAUQ_z_buf_ct{{[0-9]+}}, TAUP_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAUQ_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAUQ_z);
    //CHECK-NEXT: auto TAUP_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAUP_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::gebrd(**cusolverH, m, n, A_z_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAUQ_z_buf_ct{{[0-9]+}}, TAUP_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgebrd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZgebrd(*cusolverH, m, n, &A_z, lda, &D_d, &E_d, &TAUQ_z, &TAUP_z, &workspace_z, Lwork, &devInfo);


    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgbr_scratchpad_size<float>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgbr_scratchpad_size<float>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgbr_bufferSize(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgbr(*cusolverH, side, m, n, k, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);


    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgbr_scratchpad_size<double>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgbr_scratchpad_size<double>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::orgbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgbr_bufferSize(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgbr(*cusolverH, side, m, n, k, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungbr_scratchpad_size<std::complex<float>>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungbr_scratchpad_size<std::complex<float>>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungbr_bufferSize(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungbr(*cusolverH, side, m, n, k, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungbr_scratchpad_size<std::complex<double>>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungbr_scratchpad_size<std::complex<double>>(**cusolverH, (oneapi::mkl::generate)side, m, n, k, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::ungbr(**cusolverH, (oneapi::mkl::generate)side, m, n, k, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungbr_bufferSize(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungbr(*cusolverH, side, m, n, k, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrd_scratchpad_size<float>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrd_scratchpad_size<float>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    hipsolverDnSsytrd_bufferSize(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &Lwork);
    status = hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSsytrd(*cusolverH, uplo, n, &A_f, lda, &D_f, &E_f, &TAU_f, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::sytrd_scratchpad_size<double>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::sytrd_scratchpad_size<double>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::sytrd(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    hipsolverDnDsytrd_bufferSize(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &Lwork);
    status = hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDsytrd(*cusolverH, uplo, n, &A_d, lda, &D_d, &E_d, &TAU_d, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::hetrd_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::hetrd_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto D_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&D_f);
    //CHECK-NEXT: auto E_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&E_f);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, D_f_buf_ct{{[0-9]+}}, E_f_buf_ct{{[0-9]+}}, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    hipsolverDnChetrd_bufferSize(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &Lwork);
    status = hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnChetrd(*cusolverH, uplo, n, &A_c, lda, &D_f, &E_f, &TAU_c, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::hetrd_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::hetrd_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto D_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&D_d);
    //CHECK-NEXT: auto E_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&E_d);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::hetrd(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, D_d_buf_ct{{[0-9]+}}, E_d_buf_ct{{[0-9]+}}, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    hipsolverDnZhetrd_bufferSize(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &Lwork);
    status = hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZhetrd(*cusolverH, uplo, n, &A_z, lda, &D_d, &E_d, &TAU_z, &workspace_z, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ormtr_scratchpad_size<float>(**cusolverH, side, uplo, trans, m, n, lda, ldb));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ormtr_scratchpad_size<float>(**cusolverH, side, uplo, trans, m, n, lda, ldb);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, B_f_buf_ct{{[0-9]+}}, ldb, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto B_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&B_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, B_f_buf_ct{{[0-9]+}}, ldb, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    hipsolverDnSormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &Lwork);
    status = hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);
    hipsolverDnSormtr(*cusolverH, side, uplo, trans, m, n, &A_f, lda, &TAU_f, &B_f, ldb, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ormtr_scratchpad_size<double>(**cusolverH, side, uplo, trans, m, n, lda, ldb));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ormtr_scratchpad_size<double>(**cusolverH, side, uplo, trans, m, n, lda, ldb);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, B_d_buf_ct{{[0-9]+}}, ldb, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto B_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&B_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::ormtr(**cusolverH, side, uplo, trans, m, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, B_d_buf_ct{{[0-9]+}}, ldb, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    hipsolverDnDormtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &Lwork);
    status = hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);
    hipsolverDnDormtr(*cusolverH, side, uplo, trans, m, n, &A_d, lda, &TAU_d, &B_d, ldb, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::unmtr_scratchpad_size<std::complex<float>>(**cusolverH, side, uplo, trans, m, n, lda, ldb));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::unmtr_scratchpad_size<std::complex<float>>(**cusolverH, side, uplo, trans, m, n, lda, ldb);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, B_c_buf_ct{{[0-9]+}}, ldb, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto B_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&B_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, B_c_buf_ct{{[0-9]+}}, ldb, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    hipsolverDnCunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &Lwork);
    status = hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);
    hipsolverDnCunmtr(*cusolverH, side, uplo, trans, m, n, &A_c, lda, &TAU_c, &B_c, ldb, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::unmtr_scratchpad_size<std::complex<double>>(**cusolverH, side, uplo, trans, m, n, lda, ldb));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::unmtr_scratchpad_size<std::complex<double>>(**cusolverH, side, uplo, trans, m, n, lda, ldb);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, B_z_buf_ct{{[0-9]+}}, ldb, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto B_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&B_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::unmtr(**cusolverH, side, uplo, trans, m, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, B_z_buf_ct{{[0-9]+}}, ldb, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    hipsolverDnZunmtr_bufferSize(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &Lwork);
    status = hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);
    hipsolverDnZunmtr(*cusolverH, side, uplo, trans, m, n, &A_z, lda, &TAU_z, &B_z, ldb, &workspace_z, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgtr_scratchpad_size<float>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgtr_scratchpad_size<float>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto TAU_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&TAU_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, A_f_buf_ct{{[0-9]+}}, lda, TAU_f_buf_ct{{[0-9]+}}, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    hipsolverDnSorgtr_bufferSize(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &Lwork);
    status = hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);
    hipsolverDnSorgtr(*cusolverH, uplo, n, &A_f, lda, &TAU_f, &workspace_f, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::orgtr_scratchpad_size<double>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::orgtr_scratchpad_size<double>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto TAU_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&TAU_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::orgtr(**cusolverH, uplo, n, A_d_buf_ct{{[0-9]+}}, lda, TAU_d_buf_ct{{[0-9]+}}, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    hipsolverDnDorgtr_bufferSize(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &Lwork);
    status = hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);
    hipsolverDnDorgtr(*cusolverH, uplo, n, &A_d, lda, &TAU_d, &workspace_d, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungtr_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungtr_scratchpad_size<std::complex<float>>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto TAU_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&TAU_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, A_c_buf_ct{{[0-9]+}}, lda, TAU_c_buf_ct{{[0-9]+}}, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    hipsolverDnCungtr_bufferSize(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &Lwork);
    status = hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);
    hipsolverDnCungtr(*cusolverH, uplo, n, &A_c, lda, &TAU_c, &workspace_c, Lwork, &devInfo);

    //CHECK: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::ungtr_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda));
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::ungtr_scratchpad_size<std::complex<double>>(**cusolverH, uplo, n, lda);
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto TAU_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&TAU_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::ungtr(**cusolverH, uplo, n, A_z_buf_ct{{[0-9]+}}, lda, TAU_z_buf_ct{{[0-9]+}}, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    hipsolverDnZungtr_bufferSize(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &Lwork);
    status = hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);
    hipsolverDnZungtr(*cusolverH, uplo, n, &A_z, lda, &TAU_z, &workspace_z, Lwork, &devInfo);


    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<float>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<float>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto S_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&S_f);
    //CHECK-NEXT: auto U_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&U_f);
    //CHECK-NEXT: auto VT_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&VT_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_f_buf_ct{{[0-9]+}}, lda, S_f_buf_ct{{[0-9]+}}, U_f_buf_ct{{[0-9]+}}, ldu, VT_f_buf_ct{{[0-9]+}}, ldvt, workspace_f_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&A_f);
    //CHECK-NEXT: auto S_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&S_f);
    //CHECK-NEXT: auto U_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&U_f);
    //CHECK-NEXT: auto VT_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&VT_f);
    //CHECK-NEXT: auto workspace_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&workspace_f);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_f_buf_ct{{[0-9]+}}, lda, S_f_buf_ct{{[0-9]+}}, U_f_buf_ct{{[0-9]+}}, ldu, VT_f_buf_ct{{[0-9]+}}, ldvt, workspace_f_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnSgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);
    hipsolverDnSgesvd (*cusolverH, jobu, jobvt, m, n, &A_f, lda, &S_f, &U_f, ldu, &VT_f, ldvt, &workspace_f, Lwork, &Rwork_f, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<double>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<double>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto S_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&S_d);
    //CHECK-NEXT: auto U_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&U_d);
    //CHECK-NEXT: auto VT_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&VT_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_d_buf_ct{{[0-9]+}}, lda, S_d_buf_ct{{[0-9]+}}, U_d_buf_ct{{[0-9]+}}, ldu, VT_d_buf_ct{{[0-9]+}}, ldvt, workspace_d_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&A_d);
    //CHECK-NEXT: auto S_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&S_d);
    //CHECK-NEXT: auto U_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&U_d);
    //CHECK-NEXT: auto VT_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&VT_d);
    //CHECK-NEXT: auto workspace_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&workspace_d);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_d_buf_ct{{[0-9]+}}, lda, S_d_buf_ct{{[0-9]+}}, U_d_buf_ct{{[0-9]+}}, ldu, VT_d_buf_ct{{[0-9]+}}, ldvt, workspace_d_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnDgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);
    hipsolverDnDgesvd (*cusolverH, jobu, jobvt, m, n, &A_d, lda, &S_d, &U_d, ldu, &VT_d, ldvt, &workspace_d, Lwork, &Rwork_d, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<std::complex<float>>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<std::complex<float>>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto S_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&S_f);
    //CHECK-NEXT: auto U_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&U_c);
    //CHECK-NEXT: auto VT_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&VT_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_c_buf_ct{{[0-9]+}}, lda, S_f_buf_ct{{[0-9]+}}, U_c_buf_ct{{[0-9]+}}, ldu, VT_c_buf_ct{{[0-9]+}}, ldvt, workspace_c_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&A_c);
    //CHECK-NEXT: auto S_f_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(&S_f);
    //CHECK-NEXT: auto U_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&U_c);
    //CHECK-NEXT: auto VT_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&VT_c);
    //CHECK-NEXT: auto workspace_c_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(&workspace_c);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_c_buf_ct{{[0-9]+}}, lda, S_f_buf_ct{{[0-9]+}}, U_c_buf_ct{{[0-9]+}}, ldu, VT_c_buf_ct{{[0-9]+}}, ldvt, workspace_c_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnCgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);
    hipsolverDnCgesvd (*cusolverH, jobu, jobvt, m, n, &A_c, lda, &S_f, &U_c, ldu, &VT_c, ldvt, &workspace_c, Lwork, &Rwork_f, &devInfo);

    //CHECK: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(*(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<std::complex<double>>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobu;
    //CHECK-NEXT: oneapi::mkl::jobsvd job_ct_mkl_jobvt;
    //CHECK-NEXT: *(&Lwork) = oneapi::mkl::lapack::gesvd_scratchpad_size<std::complex<double>>(**cusolverH, job_ct_mkl_jobu, job_ct_mkl_jobvt, m, n, m, m, n);
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto S_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&S_d);
    //CHECK-NEXT: auto U_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&U_z);
    //CHECK-NEXT: auto VT_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&VT_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: status = DPCT_CHECK_ERROR(oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_z_buf_ct{{[0-9]+}}, lda, S_d_buf_ct{{[0-9]+}}, U_z_buf_ct{{[0-9]+}}, ldu, VT_z_buf_ct{{[0-9]+}}, ldvt, workspace_z_buf_ct{{[0-9]+}}, Lwork));
    //CHECK-NEXT: }
    //CHECK-NEXT: {
    //CHECK-NEXT: auto A_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&A_z);
    //CHECK-NEXT: auto S_d_buf_ct{{[0-9]+}} = dpct::get_buffer<double>(&S_d);
    //CHECK-NEXT: auto U_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&U_z);
    //CHECK-NEXT: auto VT_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&VT_z);
    //CHECK-NEXT: auto workspace_z_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<double>>(&workspace_z);
    //CHECK-NEXT: oneapi::mkl::lapack::gesvd (**cusolverH, (oneapi::mkl::jobsvd)jobu, (oneapi::mkl::jobsvd)jobvt, m, n, A_z_buf_ct{{[0-9]+}}, lda, S_d_buf_ct{{[0-9]+}}, U_z_buf_ct{{[0-9]+}}, ldu, VT_z_buf_ct{{[0-9]+}}, ldvt, workspace_z_buf_ct{{[0-9]+}}, Lwork);
    //CHECK-NEXT: }
    status = hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    hipsolverDnZgesvd_bufferSize(*cusolverH, m, n, &Lwork);
    status = hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);
    hipsolverDnZgesvd (*cusolverH, jobu, jobvt, m, n, &A_z, lda, &S_d, &U_z, ldu, &VT_z, ldvt, &workspace_z, Lwork, &Rwork_d, &devInfo);

}

