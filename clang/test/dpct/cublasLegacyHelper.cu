// RUN: dpct --no-cl-namespace-inline --format-range=none --usm-level=none -out-root %T/cublasLegacyHelper %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublasLegacyHelper/cublasLegacyHelper.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublasLegacyHelper/cublasLegacyHelper.dp.cpp -o %T/cublasLegacyHelper/cublasLegacyHelper.dp.o %}
// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <cstdio>
// CHECK: #include <dpct/blas_utils.hpp>
// CHECK: #include <complex>
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

#define cublasErrCheck(stat) { cublasErrCheck_((stat), __FILE__, __LINE__); }
void cublasErrCheck_(hipblasStatus_t stat, const char *file, int line) {
    if (stat != HIPBLAS_STATUS_SUCCESS) {
        fprintf(stderr, "cuBLAS Error: %d %s %d\n", stat, file, line);
    }
}

// CHECK: /*
// CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasInit was replaced with 0 because this functionality is redundant in SYCL.
// CHECK-NEXT: */
// CHECK-NEXT: #define MACRO_A 0
#define MACRO_A cublasInit()

#define MACRO_B(status) (status)

// CHECK: #define MACRO_C(pointer) status = DPCT_CHECK_ERROR(dpct::dpct_free(pointer))
#define MACRO_C(pointer) status = cublasFree(pointer)

void foo2(hipblasStatus_t){}

// CHECK: void foo(int, int, int, int, int, int, int, int, int, int) {}
void foo(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

// CHECK: void bar(int, int, int, int, int, int, int, int, int, int) {}
void bar(hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t, hipblasStatus_t) {}

// CHECK: int foo(int m, int n) {
hipblasStatus_t foo(int m, int n) {
  return HIPBLAS_STATUS_SUCCESS;
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK: foo(0, 1, 3, 7, 8, 11, 13, 14, 15, 16);
  foo(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);
  // CHECK: bar(0, 1, 3, 7, 8, 11, 13, 14, 15, 16);
  bar(HIPBLAS_STATUS_SUCCESS, HIPBLAS_STATUS_NOT_INITIALIZED, HIPBLAS_STATUS_ALLOC_FAILED, HIPBLAS_STATUS_INVALID_VALUE, HIPBLAS_STATUS_ARCH_MISMATCH, HIPBLAS_STATUS_MAPPING_ERROR, HIPBLAS_STATUS_EXECUTION_FAILED, HIPBLAS_STATUS_INTERNAL_ERROR, HIPBLAS_STATUS_NOT_SUPPORTED, HIPBLAS_STATUS_UNKNOWN);

  // CHECK: int status;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasInit was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasInit was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (status != 0) {
  // CHECK-NEXT:   fprintf(stderr, "!!!! CUBLAS initialization error\n");
  // CHECK-NEXT:   return EXIT_FAILURE;
  // CHECK-NEXT: }
  hipblasStatus_t status;
  status = cublasInit();
  cublasInit();
  if (status != HIPBLAS_STATUS_SUCCESS) {
    fprintf(stderr, "!!!! CUBLAS initialization error\n");
    return EXIT_FAILURE;
  }

  status = MACRO_A;

  // CHECK: int a = sizeof(int);
  // CHECK-NEXT: a = sizeof(int);
  // CHECK-NEXT: a = sizeof(dpct::queue_ptr);
  // CHECK-NEXT: a = sizeof(cl::sycl::float2);
  // CHECK-NEXT: a = sizeof(cl::sycl::double2);
  int a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasStatus_t);
  a = sizeof(hipblasHandle_t);
  a = sizeof(hipComplex);
  a = sizeof(hipDoubleComplex);

  // CHECK: dpct::queue_ptr stream1;
  // CHECK-NEXT: stream1 = dev_ct1.create_queue();
  // CHECK-NEXT: dev_ct1.set_saved_queue(stream1);
  // CHECK-NEXT: cublasErrCheck(DPCT_CHECK_ERROR(dev_ct1.set_saved_queue(stream1)));
  hipStream_t stream1;
  hipStreamCreate(&stream1);
  cublasSetKernelStream(stream1);
  cublasErrCheck(cublasSetKernelStream(stream1));

  float *d_A = NULL;
  int n = 10;
  int elemSize = 4;

  // CHECK: status = DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc((n)*(elemSize)));
  // CHECK-NEXT: d_A = (float *)dpct::dpct_malloc((n)*(elemSize));
  status = cublasAlloc(n, elemSize, (void **)&d_A);
  cublasAlloc(n, elemSize, (void **)&d_A);

  // CHECK: foo2(DPCT_CHECK_ERROR(d_A = (float *)dpct::dpct_malloc((n)*(elemSize))));
  foo2(cublasAlloc(n, elemSize, (void **)&d_A));

  // CHECK: status = DPCT_CHECK_ERROR(dpct::dpct_free(d_A));
  // CHECK-NEXT: dpct::dpct_free(d_A);
  status = cublasFree(d_A);
  cublasFree(d_A);

  // CHECK: foo2(DPCT_CHECK_ERROR(dpct::dpct_free(d_A)));
  foo2(cublasFree(d_A));

  // CHECK: MACRO_B(DPCT_CHECK_ERROR(dpct::dpct_free(d_A)));
  MACRO_B(cublasFree(d_A));

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasGetError was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: MACRO_B(0);
  MACRO_B(cublasGetError());

  MACRO_C(d_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetError was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasGetError was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = 0;
  cublasGetError();
  status = cublasGetError();

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasGetError was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo2(0);
  foo2(cublasGetError());

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasShutdown was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo2(0);
  foo2(cublasShutdown());

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasInit was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: foo2(0);
  foo2(cublasInit());

  // CHECK: /*
  // CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to cublasShutdown was replaced with 0 because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: status = 0;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasShutdown was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: return 0;
  status = cublasShutdown();
  cublasShutdown();
  return 0;
}

