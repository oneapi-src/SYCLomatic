// FIXME
// UNSUPPORTED: -windows-
// RUN: dpct --format-range=none --usm-level=none -out-root %T/USM-none %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --match-full-lines --input-file %T/USM-none/USM-none.dp.cpp %s

// CHECK: #define DPCT_USM_LEVEL_NONE
// CHECK-NEXT: #include <CL/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
#include <hip/hip_runtime.h>
#include <stdio.h>
#include <memory>

__constant__ float constData[1234567 * 4];

void foo() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;

  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMalloc((void **)&d_A, size);

  /// memcpy
  // CHECK: dpct::dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  /// memcpy async
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, stream);

  /// memcpy from symbol
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);

  /// memset
  // CHECK: dpct::dpct_memset(d_A, 23, size);
  hipMemset(d_A, 23, size);

  /// memset async
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  hipMemsetAsync(d_A, 23, size, 0);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size, *stream);
  hipMemsetAsync(d_A, 23, size, stream);

  // CHECK: h_A = (float *)malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: h_A = (float *)malloc(size);
  hipHostMalloc((void **)&h_A, size, hipHostMallocDefault);
  // CHECK: d_A = (float *)dpct::dpct_malloc(size);
  hipMallocManaged((void **)&d_A, size);

  // CHECK: h_A = (float *)malloc(size);
  hipMemAllocHost((void **)&h_A, size);

  hipDeviceptr_t* D_ptr;
  // CHECK: *D_ptr = (char *)dpct::dpct_malloc(size);
  hipMallocManaged(D_ptr, size, hipMemAttachHost);

  float* buffer[2];
#define SIZE_1 (128 * 1024 * 1024)
  // CHECK: *buffer = (float *)malloc(SIZE_1 * sizeof(float));
  // CHECK-NEXT: *(buffer + 1) = (float *)malloc(SIZE_1 * sizeof(float));
  hipHostMalloc((void**)buffer, SIZE_1 * sizeof(float), hipHostMallocDefault);
  hipHostMalloc((void**)(buffer + 1), SIZE_1 * sizeof(float), hipHostMallocDefault);
#undef SIZE_1


  // CHECK: free(h_A);
  hipHostFree(h_A);

  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of cudaHostGetDevicePointer is not supported.
  // CHECK-NEXT: */
  hipHostGetDevicePointer(&d_A, h_A, 0);

  hipHostRegister(h_A, size, 0);
  hipHostUnregister(h_A);
}

template <typename T>
int foo2() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  hipStream_t stream;
  /// memcpy from symbol
  // CHECK: dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size);
  // CHECK: dpct::dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL("constData"), size, 1);
  // CHECK: dpct::dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbol(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);

  /// memcpy from symbol async
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size);
  // CHECK: dpct::async_dpct_memcpy(h_A, constData.get_ptr(), size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 2);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 1, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 1, hipMemcpyDeviceToHost);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 2, size, dpct::device_to_host);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 2, hipMemcpyDeviceToHost, 0);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, stream);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr(*stream)) + 3, size, dpct::device_to_host, *stream);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL("constData"), size, 3, hipMemcpyDeviceToHost, stream);

  /// memcpy to symbol
  // CHECK: dpct::dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size);
  // CHECK: dpct::dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size);
  hipMemcpyToSymbol(HIP_SYMBOL("constData"), h_A, size, 1);
  // CHECK: dpct::dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);

  /// memcpy to symbol async
  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size);
  // CHECK: dpct::async_dpct_memcpy(constData.get_ptr(), h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 2);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 2, h_A, size, dpct::host_to_device);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 2, hipMemcpyHostToDevice, 0);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 3, hipMemcpyHostToDevice, stream);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr(*stream)) + 3, h_A, size, dpct::host_to_device, *stream);
  hipMemcpyToSymbolAsync(HIP_SYMBOL("constData"), h_A, size, 3, hipMemcpyHostToDevice, stream);
}

template int foo2<float>();
template int foo2<int>();

#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)

void foo3() {
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  float *d_A = NULL;
  int errorCode;
  hipPitchedPtr p_A;
  hipExtent e;
  hipMemcpy3DParms parms;
  int *data;
  size_t width, height, depth, pitch, woffset, hoffset;
  hipArray_t a1;
  int deviceID = 0;

  // CHECK: auto s1 = std::make_shared<sycl::queue *>((sycl::queue *)&q_ct1);
  // CHECK: auto s2 = std::make_shared<sycl::queue *>(&q_ct1);
  // CHECK: auto s3 = std::make_shared<sycl::queue *>(&q_ct1);
  auto s1 = std::make_shared<hipStream_t>((hipStream_t)hipStreamDefault);
  auto s2 = std::make_shared<hipStream_t>(hipStreamLegacy);
  auto s3 = std::make_shared<hipStream_t>(hipStreamPerThread);

  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, h_A, size, dpct::host_to_device), 0));
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyAsync(d_A, h_A, size, hipMemcpyHostToDevice, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device);
  // CHECK: errorCode = (dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy((char *)(constData.get_ptr()) + 1, h_A, size, dpct::host_to_device), 0));
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host);
  // CHECK: dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(h_A, (char *)(constData.get_ptr()) + 3, size, dpct::device_to_host), 0));
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpyFromSymbolAsync(h_A, HIP_SYMBOL(constData), size, 3, hipMemcpyDeviceToHost, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(d_A, size, h_A, size, size, size, dpct::host_to_device), 0));
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy);
  errorCode = hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DAsync(d_A, size, h_A, size, size, size, hipMemcpyHostToDevice, hipStreamPerThread));

  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(parms_to_data_ct1, parms_to_pos_ct1, parms_from_data_ct1, parms_from_pos_ct1, parms_size_ct1, parms_direction_ct1), 0));
  hipMemcpy3DAsync(&parms, hipStreamDefault);
  hipMemcpy3DAsync(&parms, hipStreamLegacy);
  hipMemcpy3DAsync(&parms, hipStreamPerThread);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamDefault);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamLegacy);
  errorCode = hipMemcpy3DAsync(&parms, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy3DAsync(&parms, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DFromArrayAsync(data, pitch, a1, woffset, hoffset, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, pitch, pitch, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, height, 1)), 0));
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(hipMemcpy2DToArrayAsync(a1, woffset, hoffset, data, pitch, width, height, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyToArrayAsync(a1, woffset, hoffset, data, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1));
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: errorCode = (dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memcpy(dpct::pitched_data(data, width, width, 1), sycl::id<3>(0, 0, 0), a1->to_pitched_data(), sycl::id<3>(woffset, hoffset, 0), sycl::range<3>(width, 1, 1)), 0));
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy);
  errorCode = cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread);
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamDefault));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamLegacy));
  MY_SAFE_CALL(cudaMemcpyFromArrayAsync(data, a1, woffset, hoffset, width, hipMemcpyDeviceToHost, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  // CHECK: dpct::async_dpct_memset(d_A, 23, size);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, 23, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, 23, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, 23, size), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, 23, size), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, 23, size), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, 23, size), 0));
  hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamDefault);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamLegacy);
  errorCode = hipMemsetAsync(d_A, 23, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemsetAsync(d_A, 23, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: dpct::async_dpct_memset(d_A, size, 0xf, size, size);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(d_A, size, 0xf, size, size), 0));
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy);
  errorCode = hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamDefault));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset2DAsync(d_A, size, 0xf, size, size, hipStreamPerThread));


  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: dpct::async_dpct_memset(p_A, 0xf, e);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: errorCode = (dpct::async_dpct_memset(p_A, 0xf, e), 0);
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  // CHECK: MY_SAFE_CALL((dpct::async_dpct_memset(p_A, 0xf, e), 0));
  hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy);
  errorCode = hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread);
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamDefault));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamLegacy));
  MY_SAFE_CALL(hipMemset3DAsync(p_A, 0xf, e, hipStreamPerThread));
}

/// cuda driver memory api
void foo4(){
  size_t size = 1234567 * sizeof(float);
  float *h_A = (float *)malloc(size);
  int errorCode;
  // CHECK: h_A = (float *)malloc(size);
  hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: errorCode = (h_A = (float *)malloc(size), 0);
  errorCode = hipHostAlloc((void **)&h_A, size, hipHostMallocPortable);
  // CHECK: MY_SAFE_CALL((h_A = (float *)malloc(size), 0));
  MY_SAFE_CALL(hipHostAlloc((void **)&h_A, size, hipHostMallocPortable));
}
