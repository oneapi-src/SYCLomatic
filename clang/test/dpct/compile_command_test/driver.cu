
#include <hip/hip_runtime.h>
// RUN: echo "[" > %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t.cpp\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t.cpp\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t2.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t2.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t2.cpp\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t2.cpp\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t3.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t3.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t3.cpp\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t3.cpp\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"nvcc %S/t4.cu\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t4.cu\"" >> %T/compile_commands.json
// RUN: echo "    }," >> %T/compile_commands.json
// RUN: echo "    {" >> %T/compile_commands.json
// RUN: echo "        \"command\": \"c++ %S/t4.cpp\"," >> %T/compile_commands.json
// RUN: echo "        \"directory\": \"%T\"," >> %T/compile_commands.json
// RUN: echo "        \"file\": \"%S/t4.cpp\"" >> %T/compile_commands.json
// RUN: echo "    }" >> %T/compile_commands.json
// RUN: echo "]" >> %T/compile_commands.json

// RUN: sed -i  '3,5s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '8,10s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '13,15s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '18,20s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '23,25s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '28,30s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '33,35s/\\/\//g'  %T/compile_commands.json
// RUN: sed -i  '38,40s/\\/\//g'  %T/compile_commands.json

// RUN: dpct -process-all -in-root=%S -out-root=%T -p=%T --cuda-include-path="%cuda-path/include"

// RUN: FileCheck %S/t.cpp --match-full-lines --input-file %T/t.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t.cpp -o %T/t.o %}
// RUN: FileCheck %S/t2.cpp --match-full-lines --input-file %T/t2.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t2.cpp -o %T/t2.o %}
// RUN: FileCheck %S/t3.cpp --match-full-lines --input-file %T/t3.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t3.cpp -o %T/t3.o %}
// RUN: FileCheck %S/t4.cpp --match-full-lines --input-file %T/t4.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t4.cpp -o %T/t4.o %}
// RUN: FileCheck %S/t.cu --match-full-lines --input-file %T/t.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t.dp.cpp -o %T/t.dp.o %}
// RUN: FileCheck %S/t2.cu --match-full-lines --input-file %T/t2.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t2.dp.cpp -o %T/t2.dp.o %}
// RUN: FileCheck %S/t3.cu --match-full-lines --input-file %T/t3.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t3.dp.cpp -o %T/t3.dp.o %}
// RUN: FileCheck %S/t4.cu --match-full-lines --input-file %T/t4.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/t4.dp.cpp -o %T/t4.dp.o %}
