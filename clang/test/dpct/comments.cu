#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-12.0, cuda-12.1, cuda-12.2, cuda-12.3, cuda-12.4, cuda-12.5, cuda-12.6
// UNSUPPORTED: v12.0, v12.1, v12.2, v12.3, v12.4, v12.5, v12.6
// RUN: dpct --format-range=none -out-root %T/comments %s --cuda-include-path="%cuda-path/include" --comments -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/comments/comments.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/comments/comments.dp.cpp -o %T/comments/comments.dp.o %}

static texture<uint2, 1> tex21;

__constant__ int a = 1;
__device__ int b[36][36];

__device__ void test() {
  __shared__ int cl[36];
  cl[0] = b[0][0] + a;
}

__device__ uint2 al[16];
__global__ void kernel() {
  test();
  __shared__ int bl[12][12];
  al[0] = tex1D(tex21, 1);
  bl[0][0] = 0;
  printf("test\n");
}

int main() {
    dim3 griddim(1, 2, 3);
    dim3 threaddim(1, 2, 3);

//      CHECK:    {
// CHECK-NEXT:      // init global memory
// CHECK-NEXT:      a.init();
// CHECK-NEXT:      b.init();
// CHECK-NEXT:      al.init();
// CHECK-EMPTY:
// CHECK:           tex21.create_image();
// CHECK-NEXT:      dpct::get_in_order_queue().submit(
// CHECK-NEXT:          [&](sycl::handler &cgh) {
// CHECK-NEXT:            sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:            // pointers to device memory
// CHECK-NEXT:            auto a_ptr_ct1 = a.get_ptr();
// CHECK-NEXT:            auto al_ptr_ct1 = al.get_ptr();
// CHECK-EMPTY:  
// CHECK-NEXT:            // accessors to device memory
// CHECK-NEXT:            sycl::local_accessor<int, 1> cl_acc_ct1(sycl::range<1>(36), cgh);
// CHECK-NEXT:            sycl::local_accessor<int[12][12], 0> bl_acc_ct1(cgh);
// CHECK-NEXT:            auto b_acc_ct1 = b.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:            // accessors to image objects
// CHECK-NEXT:            auto tex21_acc = tex21.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:            // sampler of image objects
// CHECK-NEXT:            auto tex21_smpl = tex21.get_sampler();
// CHECK-EMPTY:  
// CHECK-NEXT:            cgh.parallel_for(
// CHECK-NEXT:              sycl::nd_range<3>(griddim * threaddim, threaddim),
// CHECK-NEXT:              [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:                kernel(stream_ct1, *a_ptr_ct1, b_acc_ct1, al_ptr_ct1, cl_acc_ct1.get_multi_ptr<sycl::access::decorated::no>().get(), bl_acc_ct1, dpct::image_accessor_ext<sycl::uint2, 1>(tex21_smpl, tex21_acc));
// CHECK-NEXT:              });
// CHECK-NEXT:          });
// CHECK-NEXT:    }
    kernel<<<griddim, threaddim>>>();
}

