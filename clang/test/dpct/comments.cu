#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-12.0
// UNSUPPORTED: v12.0
// RUN: dpct --format-range=none -out-root %T/comments %s --cuda-include-path="%cuda-path/include" --comments -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/comments/comments.dp.cpp

static texture<uint2, 1> tex21;

__constant__ int a = 1;
__device__ int b[36][36];

__device__ void test() {
  __shared__ int cl[36];
  cl[0] = b[0][0] + a;
}

__device__ uint2 al[16];
__global__ void kernel() {
  test();
  __shared__ int bl[12][12];
  al[0] = tex1D(tex21, 1);
  bl[0][0] = 0;
  printf("test\n");
}

int main() {
    // CHECK: // These variables are defined for 3d matrix memory copy.
    // CHECK-NEXT: dpct::pitched_data p_from_data_ct1, p_to_data_ct1;
    // CHECK-NEXT: sycl::id<3> p_from_pos_ct1(0, 0, 0), p_to_pos_ct1(0, 0, 0);
    // CHECK-NEXT: sycl::range<3> p_size_ct1(1, 1, 1);
    // CHECK-NEXT: dpct::memcpy_direction p_direction_ct1;
    hipMemcpy3DParms p;
    dim3 griddim(1, 2, 3);
    dim3 threaddim(1, 2, 3);

// CHECK:    dpct::get_default_queue().submit(
// CHECK-NEXT:        [&](sycl::handler &cgh) {
// CHECK-NEXT:          sycl::stream stream_ct1(64 * 1024, 80, cgh);
// CHECK-EMPTY:
// CHECK-NEXT:          // init global memory
// CHECK-NEXT:          a.init();
// CHECK-NEXT:          b.init();
// CHECK-NEXT:          al.init();
// CHECK-EMPTY:  
// CHECK-NEXT:          // pointers to device memory
// CHECK-NEXT:          auto a_ptr_ct1 = a.get_ptr();
// CHECK-NEXT:          auto al_ptr_ct1 = al.get_ptr();
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to device memory
// CHECK-NEXT:          sycl::local_accessor<int, 1> cl_acc_ct1(sycl::range<1>(36), cgh);
// CHECK-NEXT:          sycl::local_accessor<int, 2> bl_acc_ct1(sycl::range<2>(12, 12), cgh);
// CHECK-NEXT:          auto b_acc_ct1 = b.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // accessors to image objects
// CHECK-NEXT:          auto tex21_acc = tex21.get_access(cgh);
// CHECK-EMPTY:  
// CHECK-NEXT:          // sampler of image objects
// CHECK-NEXT:          auto tex21_smpl = tex21.get_sampler();
// CHECK-EMPTY:  
// CHECK-NEXT:          cgh.parallel_for(
// CHECK-NEXT:            sycl::nd_range<3>(griddim * threaddim, threaddim),
// CHECK-NEXT:            [=](sycl::nd_item<3> item_ct1) {
// CHECK-NEXT:              kernel(stream_ct1, *a_ptr_ct1, b_acc_ct1, al_ptr_ct1, cl_acc_ct1.get_pointer(), bl_acc_ct1, dpct::image_accessor_ext<sycl::uint2, 1>(tex21_smpl, tex21_acc));
// CHECK-NEXT:            });
// CHECK-NEXT:        });
    kernel<<<griddim, threaddim>>>();
}

