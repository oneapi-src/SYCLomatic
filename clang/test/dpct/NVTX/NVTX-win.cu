// UNSUPPORTED: system-linux
// RUN: dpct --format-range=none -in-root %S -out-root %T %S/NVTX-win.cu --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/NVTX-win.dp.cpp --match-full-lines %s

// CHECK: #include <sycl/sycl.hpp>
// CHECK: #include <dpct/dpct.hpp>
#include "nvToolsExtCuda.h"

int main(){
  hipDevice_t* device;
  hipDeviceGet(device,0);
  // CHECK:     /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of nvtxNameCuDeviceW is not supported.
  // CHECK-NEXT: */
  nvtxNameCuDeviceW(*device,"nvtx_device");
}
