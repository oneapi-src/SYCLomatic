
#include <hip/hip_runtime.h>
// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5
// RUN: dpct --format-range=none -out-root %T/api_mapping_cases/Runtime %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14 2>&1 | FileCheck %s
// CHECK-NOT: {{.*}}error{{.*}}

void test(hipEvent_t e) { hipEventDestroy(e /*cudaEvent_t*/); }
