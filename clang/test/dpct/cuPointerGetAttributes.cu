// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none -out-root %T/Out/cuPointerGetAttributes %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/Out/cuPointerGetAttributes/cuPointerGetAttributes.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/Out/cuPointerGetAttributes/cuPointerGetAttributes.dp.cpp -o %T/Out/cuPointerGetAttributes/cuPointerGetAttributes.dp.o %}
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>

int main() {
  int N = 2048;
  size_t size = N * sizeof(float);

  float *h_A = (float *)malloc(size);

  float *d_A;
  hipMalloc((void **)&d_A, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);

  unsigned int numAttributes = 5;

  // CHECK: dpct::pointer_attributes::type attributes[] = {
  // CHECK:   dpct::pointer_attributes::type::memory_type,
  // CHECK:   dpct::pointer_attributes::type::device_pointer,
  // CHECK:   dpct::pointer_attributes::type::host_pointer,
  // CHECK:   dpct::pointer_attributes::type::is_managed,
  // CHECK:   dpct::pointer_attributes::type::device_id
  hipPointer_attribute attributes[] = {
    HIP_POINTER_ATTRIBUTE_MEMORY_TYPE,
    HIP_POINTER_ATTRIBUTE_DEVICE_POINTER,
    HIP_POINTER_ATTRIBUTE_HOST_POINTER,
    HIP_POINTER_ATTRIBUTE_IS_MANAGED,
    HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL
  };

  // CHECK: dpct::pointer_attributes::type attributes_unsupported[] = {
  // CHECK:   dpct::pointer_attributes::type::unsupported,
  // CHECK:   dpct::pointer_attributes::type::unsupported
  hipPointer_attribute attributes_unsupported[] = {
    HIP_POINTER_ATTRIBUTE_CONTEXT,
    HIP_POINTER_ATTRIBUTE_IS_LEGACY_HIP_IPC_CAPABLE
  };

  // CHECK: sycl::usm::alloc memType;
  hipMemoryType memType;
  void* hostPtr;
  unsigned int isManaged;
  int deviceID;
  // CHECK: dpct::device_ptr devPtr;
  hipDeviceptr_t devPtr;
  hipCtx_t cuCtx;
  bool is_legacy_cuda_ipc_capable;

  void* attributeValues[] = {
    &memType,
    &devPtr,
    &hostPtr,
    &isManaged,
    &deviceID
  };

  void* attributeValues_unsupported[] = {
    &cuCtx,
    &is_legacy_cuda_ipc_capable
  };

  // CHECK: dpct::pointer_attributes::get(numAttributes, attributes, attributeValues, (dpct::device_ptr) h_A);
  hipDrvPointerGetAttributes(
    numAttributes,
    attributes,
    attributeValues,
    (hipDeviceptr_t) h_A
  );

  std::cout << "====== Host Attributes =======" << std::endl;
  std::cout << deviceID << std::endl;
  std::cout << static_cast<int>(memType) << std::endl;
  std::cout << hostPtr << std::endl;
  std::cout << devPtr << std::endl;
  std::cout << isManaged << std::endl;

  void * malloc_host;
  hipHostMalloc((void **)&malloc_host, size, hipHostMallocDefault);
  // CHECK: dpct::pointer_attributes::get(numAttributes, attributes, attributeValues, (dpct::device_ptr) malloc_host);
  hipDrvPointerGetAttributes(
    numAttributes,
    attributes,
    attributeValues,
    (hipDeviceptr_t) malloc_host
  );
  std::cout << "====== Malloc Host Attributes =======" << std::endl;
  std::cout << "malloc host " << malloc_host << std::endl;
  std::cout << deviceID << std::endl;
  std::cout << static_cast<int>(memType) << std::endl;
  std::cout << hostPtr << std::endl;
  std::cout << devPtr << std::endl;
  std::cout << isManaged << std::endl;

  // CHECK: dpct::pointer_attributes::get(numAttributes, attributes, attributeValues, (dpct::device_ptr) d_A);
  hipDrvPointerGetAttributes(
    numAttributes,
    attributes,
    attributeValues,
    (hipDeviceptr_t) d_A
  );
  std::cout << "====== Device Attributes =======" << std::endl;
  std::cout << *static_cast<int *>(attributeValues[0]) << std::endl;
  std::cout << attributeValues[1] << std::endl;
  std::cout << attributeValues[2] << std::endl;
  std::cout << *static_cast<unsigned int *>(attributeValues[3]) << std::endl;
  std::cout << *static_cast<int *>(attributeValues[4]) << std::endl;

  // CHECK: if (memType == sycl::usm::alloc::host) {
  if (memType == hipMemoryTypeHost) {
    return 0;
  // CHECK: } else if (memType == sycl::usm::alloc::device) {
  } else if (memType == hipMemoryTypeDevice) {
    return 1;
  } else if (isManaged) {
    return 2;
  }

  // CHECK: dpct::pointer_attributes::get(2, attributes_unsupported, attributeValues_unsupported, (dpct::device_ptr) d_A);
  hipDrvPointerGetAttributes(
    2,
    attributes_unsupported,
    attributeValues_unsupported,
    (hipDeviceptr_t) d_A
  );
}
