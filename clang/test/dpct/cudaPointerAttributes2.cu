// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2
// RUN: dpct --format-range=none -out-root %T/Out/cudaPointerAttributes2 %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck %s --match-full-lines --input-file %T/Out/cudaPointerAttributes2/cudaPointerAttributes2.dp.cpp
#include <hip/hip_runtime.h>

#include <stdlib.h>
#include <iostream>
int main() {
  int N = 2048;
  size_t size = N * sizeof(float);

  float *h_A = (float *)malloc(size);
  float *h_B = (float *)malloc(size);

  float *d_A;
  float *d_B;
  float *d_C;
  hipMalloc((void **)&d_A, size);
  hipMalloc((void **)&d_B, size);
  hipMalloc((void **)&d_C, size);

  hipMemcpy(d_A, h_A, size, hipMemcpyHostToDevice);
  hipMemcpy(d_B, h_B, size, hipMemcpyHostToDevice);
  // CHECK: dpct::pointer_attributes attributes;
  hipPointerAttribute_t attributes;
  // CHECK: attributes.init(h_A);
  hipPointerGetAttributes(&attributes, h_A);
  // CHECK: if (attributes.get_device_id() != (unsigned int)-1) {
  // CHECK: attributes.get_memory_type();
  // CHECK: std::cout << attributes.get_host_pointer() << std::endl;
  // CHECK: std::cout << attributes.get_device_pointer() << std::endl;
  std::cout << "====== Host Attributes =======" << std::endl;
  // The malloc memory is not a USM allocation memory. So the device attr is the default value(-1);
  if (attributes.device != (unsigned int)-1) {
    return -1;
  }
  attributes.type;
  std::cout << attributes.hostPointer << std::endl;
  std::cout << attributes.devicePointer << std::endl;

  void * malloc_host;
  hipHostMalloc((void **)&malloc_host, size, hipHostMallocDefault);
  hipPointerAttribute_t attributes2;
  hipPointerGetAttributes (&attributes2, malloc_host);
  std::cout << "====== Malloc Host Attributes =======" << std::endl;
  std::cout << "malloc host " << malloc_host << std::endl;
  std::cout << attributes2.device << std::endl;
  attributes2.type;
  std::cout << attributes2.hostPointer << std::endl;
  std::cout << attributes2.devicePointer << std::endl;

  // CHECK: dpct::pointer_attributes *attributes3 = new dpct::pointer_attributes();
  hipPointerAttribute_t *attributes3 = new hipPointerAttribute_t();
  // CHECK: attributes3->init(d_A);
  hipPointerGetAttributes (attributes3, d_A);
  // CHECK: std::cout << attributes3->get_device_id() << std::endl;
  // CHECK: attributes3->get_memory_type();
  std::cout << "====== Device Attributes =======" << std::endl;
  std::cout << attributes3->device << std::endl;
  attributes3->type;
  std::cout << attributes3->hostPointer << std::endl;
  std::cout << attributes3->devicePointer << std::endl;
  // CHECK: if (attributes3->get_memory_type() == sycl::usm::alloc::host) {
  // CHECK: } else if (attributes3->get_memory_type() == sycl::usm::alloc::device) {
  // CHECK: } else if (attributes3->get_memory_type() == sycl::usm::alloc::shared) {
  // CHECK: } else if (attributes3->get_memory_type() == sycl::usm::alloc::unknown) {
  if (attributes3->type == hipMemoryTypeHost) {
    return 0;
  } else if (attributes3->type == hipMemoryTypeDevice) {
    return 1;
  } else if (attributes3->type == hipMemoryTypeManaged) {
    return 2;
  } else if (attributes3->type == cudaMemoryTypeUnregistered) {
    return 3;
  }
}
