//RUN: dpct --out-root %T/curandEnum --format-range=none --cuda-include-path="%cuda-path/include" %s -- -x cuda --cuda-host-only
//RUN: FileCheck --input-file %T/curandEnum/curandEnum.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/curandEnum/curandEnum.dp.cpp -o %T/curandEnum/curandEnum.dp.o %}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

// CHECK:int foo(
// CHECK-NEXT:int a1,
// CHECK-NEXT:int a2,
// CHECK-NEXT:int a3,
// CHECK-NEXT:int a4,
// CHECK-NEXT:int a5,
// CHECK-NEXT:int a6,
// CHECK-NEXT:int a7,
// CHECK-NEXT:int a8,
// CHECK-NEXT:int a9,
// CHECK-NEXT:int a10,
// CHECK-NEXT:int a11,
// CHECK-NEXT:int a12,
// CHECK-NEXT:int a13) {}
hiprandStatus_t foo(
  hiprandStatus_t a1,
  hiprandStatus_t a2,
  hiprandStatus_t a3,
  hiprandStatus_t a4,
  hiprandStatus_t a5,
  hiprandStatus_t a6,
  hiprandStatus_t a7,
  hiprandStatus_t a8,
  hiprandStatus_t a9,
  hiprandStatus_t a10,
  hiprandStatus_t a11,
  hiprandStatus_t a12,
  hiprandStatus_t a13) {}

int main() {
  // CHECK:int a1 = 0;
  // CHECK-NEXT:int a2 = 100;
  // CHECK-NEXT:int a3 = 101;
  // CHECK-NEXT:int a4 = 102;
  // CHECK-NEXT:int a5 = 103;
  // CHECK-NEXT:int a6 = 104;
  // CHECK-NEXT:int a7 = 105;
  // CHECK-NEXT:int a8 = 106;
  // CHECK-NEXT:int a9 = 201;
  // CHECK-NEXT:int a10 = 202;
  // CHECK-NEXT:int a11 = 203;
  // CHECK-NEXT:int a12 = 204;
  // CHECK-NEXT:int a13 = 999;
  hiprandStatus_t a1 = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t a2 = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t a3 = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t a4 = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t a5 = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t a6 = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t a7 = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t a8 = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t a9 = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t a10 = HIPRAND_STATUS_PREEXISTING_FAILURE;
  hiprandStatus_t a11 = HIPRAND_STATUS_INITIALIZATION_FAILED;
  hiprandStatus_t a12 = HIPRAND_STATUS_ARCH_MISMATCH;
  hiprandStatus_t a13 = HIPRAND_STATUS_INTERNAL_ERROR;


  // CHECK:foo(
  // CHECK-NEXT:  0,
  // CHECK-NEXT:  100,
  // CHECK-NEXT:  101,
  // CHECK-NEXT:  102,
  // CHECK-NEXT:  103,
  // CHECK-NEXT:  104,
  // CHECK-NEXT:  105,
  // CHECK-NEXT:  106,
  // CHECK-NEXT:  201,
  // CHECK-NEXT:  202,
  // CHECK-NEXT:  203,
  // CHECK-NEXT:  204,
  // CHECK-NEXT:  999);
  foo(
    HIPRAND_STATUS_SUCCESS,
    HIPRAND_STATUS_VERSION_MISMATCH,
    HIPRAND_STATUS_NOT_INITIALIZED,
    HIPRAND_STATUS_ALLOCATION_FAILED,
    HIPRAND_STATUS_TYPE_ERROR,
    HIPRAND_STATUS_OUT_OF_RANGE,
    HIPRAND_STATUS_LENGTH_NOT_MULTIPLE,
    HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED,
    HIPRAND_STATUS_LAUNCH_FAILURE,
    HIPRAND_STATUS_PREEXISTING_FAILURE,
    HIPRAND_STATUS_INITIALIZATION_FAILED,
    HIPRAND_STATUS_ARCH_MISMATCH,
    HIPRAND_STATUS_INTERNAL_ERROR);
}

