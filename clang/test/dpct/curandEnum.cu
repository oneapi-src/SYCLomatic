// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
//RUN: dpct --out-root %T/curandEnum --format-range=none --cuda-include-path="%cuda-path/include" %s -- -x cuda --cuda-host-only
//RUN: FileCheck --input-file %T/curandEnum/curandEnum.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/curandEnum/curandEnum.dp.cpp -o %T/curandEnum/curandEnum.dp.o %}

#include <hip/hip_runtime.h>
#include <stdio.h>
#include <hiprand/hiprand.h>

// CHECK:int foo(
// CHECK-NEXT:int a1,
// CHECK-NEXT:int a2,
// CHECK-NEXT:int a3,
// CHECK-NEXT:int a4,
// CHECK-NEXT:int a5,
// CHECK-NEXT:int a6,
// CHECK-NEXT:int a7,
// CHECK-NEXT:int a8,
// CHECK-NEXT:int a9,
// CHECK-NEXT:int a10,
// CHECK-NEXT:int a11,
// CHECK-NEXT:int a12,
// CHECK-NEXT:int a13) {}
hiprandStatus_t foo(
  hiprandStatus_t a1,
  hiprandStatus_t a2,
  hiprandStatus_t a3,
  hiprandStatus_t a4,
  hiprandStatus_t a5,
  hiprandStatus_t a6,
  hiprandStatus_t a7,
  hiprandStatus_t a8,
  hiprandStatus_t a9,
  hiprandStatus_t a10,
  hiprandStatus_t a11,
  hiprandStatus_t a12,
  hiprandStatus_t a13) {}

// CHECK:uint32_t goo(
// CHECK-NEXT:uint32_t b1,
// CHECK-NEXT:uint32_t b2,
// CHECK-NEXT:// curandOrdering_t b3,
// CHECK-NEXT:uint32_t b4,
// CHECK-NEXT:uint32_t b5
// CHECK-NEXT:// , curandOrdering_t b6
// CHECK-NEXT:) { return b1; }
hiprandOrdering_t goo(
    hiprandOrdering_t b1,
    hiprandOrdering_t b2,
    // curandOrdering_t b3,
    hiprandOrdering_t b4,
    hiprandOrdering_t b5
    // , curandOrdering_t b6
) { return b1; }

int main() {
  // CHECK:int a1 = 0;
  // CHECK-NEXT:int a2 = 100;
  // CHECK-NEXT:int a3 = 101;
  // CHECK-NEXT:int a4 = 102;
  // CHECK-NEXT:int a5 = 103;
  // CHECK-NEXT:int a6 = 104;
  // CHECK-NEXT:int a7 = 105;
  // CHECK-NEXT:int a8 = 106;
  // CHECK-NEXT:int a9 = 201;
  // CHECK-NEXT:int a10 = 202;
  // CHECK-NEXT:int a11 = 203;
  // CHECK-NEXT:int a12 = 204;
  // CHECK-NEXT:int a13 = 999;
  hiprandStatus_t a1 = HIPRAND_STATUS_SUCCESS;
  hiprandStatus_t a2 = HIPRAND_STATUS_VERSION_MISMATCH;
  hiprandStatus_t a3 = HIPRAND_STATUS_NOT_INITIALIZED;
  hiprandStatus_t a4 = HIPRAND_STATUS_ALLOCATION_FAILED;
  hiprandStatus_t a5 = HIPRAND_STATUS_TYPE_ERROR;
  hiprandStatus_t a6 = HIPRAND_STATUS_OUT_OF_RANGE;
  hiprandStatus_t a7 = HIPRAND_STATUS_LENGTH_NOT_MULTIPLE;
  hiprandStatus_t a8 = HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED;
  hiprandStatus_t a9 = HIPRAND_STATUS_LAUNCH_FAILURE;
  hiprandStatus_t a10 = HIPRAND_STATUS_PREEXISTING_FAILURE;
  hiprandStatus_t a11 = HIPRAND_STATUS_INITIALIZATION_FAILED;
  hiprandStatus_t a12 = HIPRAND_STATUS_ARCH_MISMATCH;
  hiprandStatus_t a13 = HIPRAND_STATUS_INTERNAL_ERROR;


  // CHECK:foo(
  // CHECK-NEXT:  0,
  // CHECK-NEXT:  100,
  // CHECK-NEXT:  101,
  // CHECK-NEXT:  102,
  // CHECK-NEXT:  103,
  // CHECK-NEXT:  104,
  // CHECK-NEXT:  105,
  // CHECK-NEXT:  106,
  // CHECK-NEXT:  201,
  // CHECK-NEXT:  202,
  // CHECK-NEXT:  203,
  // CHECK-NEXT:  204,
  // CHECK-NEXT:  999);
  foo(
    HIPRAND_STATUS_SUCCESS,
    HIPRAND_STATUS_VERSION_MISMATCH,
    HIPRAND_STATUS_NOT_INITIALIZED,
    HIPRAND_STATUS_ALLOCATION_FAILED,
    HIPRAND_STATUS_TYPE_ERROR,
    HIPRAND_STATUS_OUT_OF_RANGE,
    HIPRAND_STATUS_LENGTH_NOT_MULTIPLE,
    HIPRAND_STATUS_DOUBLE_PRECISION_REQUIRED,
    HIPRAND_STATUS_LAUNCH_FAILURE,
    HIPRAND_STATUS_PREEXISTING_FAILURE,
    HIPRAND_STATUS_INITIALIZATION_FAILED,
    HIPRAND_STATUS_ARCH_MISMATCH,
    HIPRAND_STATUS_INTERNAL_ERROR);

  // CHECK:uint32_t b1 = 81920;
  // CHECK-NEXT:uint32_t b2 = 81920;
  // CHECK-NEXT:// curandOrdering_t b3 = CURAND_ORDERING_PSEUDO_SEEDED;
  // CHECK-NEXT:uint32_t b4 = 4096;
  // CHECK-NEXT:uint32_t b5 = 0;
  // CHECK-NEXT:// curandOrdering_t b6 = CURAND_ORDERING_QUASI_DEFAULT;
  hiprandOrdering_t b1 = HIPRAND_ORDERING_PSEUDO_BEST;
  hiprandOrdering_t b2 = HIPRAND_ORDERING_PSEUDO_DEFAULT;
  // curandOrdering_t b3 = CURAND_ORDERING_PSEUDO_SEEDED;
  hiprandOrdering_t b4 = HIPRAND_ORDERING_PSEUDO_LEGACY;
  hiprandOrdering_t b5 = HIPRAND_ORDERING_PSEUDO_DYNAMIC;
  // curandOrdering_t b6 = CURAND_ORDERING_QUASI_DEFAULT;

  // CHECK:goo(
  // CHECK-NEXT:  81920,
  // CHECK-NEXT:  81920,
  // CHECK-NEXT:  // CURAND_ORDERING_PSEUDO_SEEDED,
  // CHECK-NEXT:  4096,
  // CHECK-NEXT:  0
  // CHECK-NEXT:  // , CURAND_ORDERING_QUASI_DEFAULT
  // CHECK-NEXT:);
  goo(
      HIPRAND_ORDERING_PSEUDO_BEST,
      HIPRAND_ORDERING_PSEUDO_DEFAULT,
      // CURAND_ORDERING_PSEUDO_SEEDED,
      HIPRAND_ORDERING_PSEUDO_LEGACY,
      HIPRAND_ORDERING_PSEUDO_DYNAMIC
      // , CURAND_ORDERING_QUASI_DEFAULT
  );
}
