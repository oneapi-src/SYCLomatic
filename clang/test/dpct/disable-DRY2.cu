// RUN: dpct --format-range=none -out-root %T/disable-DRY2 %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/disable-DRY2/disable-DRY2.dp.cpp


#include "hip/hip_runtime.h"

void bar(){
  int device = 0;
// CHECK:/*
// CHECK-NEXT:DPCT1093:{{[0-9]+}}: The "device" device may be not the one intended for use. Adjust the selected device if needed.
// CHECK-NEXT:*/
//CHECK-NEXT:dpct::select_device(device);
  hipSetDevice(device);
}

#define SIZE 100

size_t size = 1234567 * sizeof(float);
float *h_A = (float *)malloc(size);
float *d_A = NULL;
__constant__ float constData[123 * 4];
hipStream_t s;

// CHECK: void foo1() {
// CHECK-NEXT: dpct::get_in_order_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE );
// CHECK-NEXT: dpct::get_in_order_queue().memcpy( d_A, h_A, sizeof(double)*SIZE*SIZE ).wait();
// CHECK-NEXT: dpct::get_in_order_queue().memcpy((char *)(constData.get_ptr()) + 1, h_A, size).wait();
// CHECK-NEXT: dpct::get_in_order_queue().memset(d_A, 23, size).wait();
// CHECK-NEXT: dpct::get_in_order_queue().memset(d_A, 23, size).wait();
// CHECK-NEXT: bar();
// CHECK-NEXT: dpct::get_in_order_queue().memset(d_A, 23, size).wait();
// CHECK-NEXT: dpct::get_in_order_queue().memset(d_A, 23, size).wait();
// CHECK-NEXT: }
void foo1() {
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpy( d_A, h_A, sizeof(double)*SIZE*SIZE, hipMemcpyDeviceToHost );
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1);
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
  bar();
  hipMemset(d_A, 23, size);
  hipMemset(d_A, 23, size);
}

