// RUN: dpct --format-range=none -out-root %T/driver_device %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/driver_device/driver_device.dp.cpp
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

#define NUM 1
#define MY_SAFE_CALL(CALL) do {    \
  int Error = CALL;                \
} while (0)
int main(){
  int result1, result2;

  int *presult1 = &result1, *presult2 = &result2;
  size_t size;
  // CHECK: int device;
  hipDevice_t device;

  // CHECK: int *pdevice = &device;
  hipDevice_t *pdevice = &device;

  // CHECK: device = 0;
  hipDeviceGet(&device, 0);

  // CHECK: device = NUM;
  hipDeviceGet(&device, NUM);

  // CHECK: *pdevice = 0;
  hipDeviceGet(pdevice, 0);

  // CHECK: *((int *)pdevice) = 0;
  hipDeviceGet((hipDevice_t *)pdevice, 0);

  // CHECK: MY_SAFE_CALL((device = 0, 0));
  MY_SAFE_CALL(hipDeviceGet(&device, 0));

  // CHECK: /*
  // CHECK-NEXT: DPCT1076:{{[0-9]+}}: The device attribute was not recognized. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: hipDeviceGetAttribute(&result1, attr, device);
  hipDeviceAttribute_t attr = hipDeviceAttributeMaxThreadsPerBlock;
  hipDeviceGetAttribute(&result1, attr, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMajor, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeComputeCapabilityMinor, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_integrated();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeIntegrated, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_max_clock_frequency();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeClockRate, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_max_compute_units();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).is_native_atomic_supported();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeHostNativeAtomicSupported, device);

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_work_item_max_dim_x_size();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxBlockDimX, device);
  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_local_mem_size();
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxSharedMemoryPerBlock, device);
  // CHECK: int context;
  hipCtx_t context;
  // CHECK: unsigned int flags = 0;
  unsigned int flags = hipDeviceMapHost;
  // CHECK: flags += 0;
  flags += hipDeviceScheduleBlockingSync;
  // CHECK: flags += 0;
  flags += hipDeviceScheduleSpin;
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  if (hipCtxCreate(&context, flags, device) == hipSuccess) {
    return 0;
  }
  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipCtxSetCacheConfig was removed because SYCL currently does not support setting cache config on devices.
  // CHECK-NEXT: */
  hipCtxSetCacheConfig(hipFuncCachePreferShared);

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceSetLimit was removed because SYCL currently does not support setting resource limits on devices.
  // CHECK-NEXT: */
  hipDeviceSetLimit(hipLimitPrintfFifoSize, 10);
  size_t printfsize;

  // CHECK: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipDeviceGetLimit was removed because SYCL currently does not support getting resource limits on devices.
  // CHECK-NEXT: */
  hipDeviceGetLimit(&printfsize, hipLimitPrintfFifoSize);


  // CHECK: /*
  // CHECK-NEXT: DPCT1007:{{[0-9]+}}: Migration of hipDeviceGetAttribute is not supported.
  // CHECK-NEXT: */
  hipDeviceGetAttribute(&result1, hipDeviceAttributeMaxThreadsPerBlock, device);

  // CHECK: MY_SAFE_CALL((result1 = dpct::dev_mgr::instance().get_device(device).get_max_compute_units(), 0));
  MY_SAFE_CALL(hipDeviceGetAttribute(&result1, hipDeviceAttributeMultiprocessorCount, device));

  // CHECK: result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  // CHECK: result2 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  hipDeviceComputeCapability(&result1, &result2, device);

  // CHECK: MY_SAFE_CALL([&](){
  // CHECK-NEXT:   result1 = dpct::dev_mgr::instance().get_device(device).get_major_version();
  // CHECK-NEXT:   result2 = dpct::dev_mgr::instance().get_device(device).get_minor_version();
  // CHECK-NEXT:   return 0;
  // CHECK-NEXT: }());
  MY_SAFE_CALL(hipDeviceComputeCapability(&result1, &result2, device));

  // CHECK: /*
  // CHECK-NEXT: DPCT1043:{{[0-9]+}}: The version-related API is different in SYCL. An initial code was generated, but you need to adjust it.
  // CHECK-NEXT: */
  // CHECK-NEXT: result1 = std::stoi(dpct::get_current_device().get_info<sycl::info::device::version>());
  hipDriverGetVersion(&result1);

  // CHECK: MY_SAFE_CALL((result1 = dpct::dev_mgr::instance().device_count(), 0));
  MY_SAFE_CALL(hipGetDeviceCount(&result1));

  // CHECK: result1 = dpct::dev_mgr::instance().device_count();
  hipGetDeviceCount(&result1);

  // CHECK: MY_SAFE_CALL((result1 = dpct::dev_mgr::instance().device_count(), 0));
  MY_SAFE_CALL(hipGetDeviceCount(&result1));

  char name[100];

  // CHECK: memcpy(name, dpct::dev_mgr::instance().get_device(device).get_info<sycl::info::device::name>().c_str(), 90);
  hipDeviceGetName(name, 90, device);

  // CHECK: MY_SAFE_CALL((memcpy(name, dpct::dev_mgr::instance().get_device(device).get_info<sycl::info::device::name>().c_str(), 90), 0));
  MY_SAFE_CALL(hipDeviceGetName(name, 90, device));
  // CHECK: size = dpct::dev_mgr::instance().get_device(device).get_device_info().get_global_mem_size();
  hipDeviceTotalMem(&size, device);

  return 0;
}
