#include "hip/hip_runtime.h"
// RUN: dpct -out-root %T/types001 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/types001/types001.dp.cpp

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <vector>
#include <algorithm>

// CHECK: dpct::device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const dpct::device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile dpct::device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHDCK: dpct::event_ptr events[23];
hipEvent_t events[23];
// CHECK: const dpct::event_ptr *pevents[23];
const hipEvent_t *pevents[23];
// CHECK: const dpct::event_ptr **ppevents[23];
const hipEvent_t **ppevents[23];

// CHECK: dpct::err0 errors[23];
hipError_t errors[23];
// CHECK: const dpct::err0 *perrors[23];
const hipError_t *perrors[23];
// CHECK: const dpct::err0 **pperrors[23];
const hipError_t **pperrors[23];

// CHECK: dpct::err0 errors1[23];
hipError_t errors1[23];
// CHECK: const dpct::err0 *perrors1[23];
const hipError_t *perrors1[23];
// CHECK: const dpct::err0 **pperrors1[23];
const hipError_t **pperrors1[23];

// CHECK: sycl::range<3> dims[23];
dim3 dims[23];
// CHECK: const sycl::range<3> *pdims[23];
const dim3 *pdims[23];
// CHECK: const sycl::range<3> **ppdims[23];
const dim3 **ppdims[23];

struct s {
  // CHECK: dpct::event_ptr events[23];
  hipEvent_t events[23];
  // CHECK: const dpct::event_ptr *pevents[23];
  const hipEvent_t *pevents[23];
  // CHECK: const dpct::event_ptr **ppevents[23];
  const hipEvent_t **ppevents[23];

  // CHECK: dpct::err0 errors[23];
  hipError_t errors[23];
  // CHECK: const dpct::err0 *perrors[23];
  const hipError_t *perrors[23];
  // CHECK: const dpct::err0 **pperrors[23];
  const hipError_t **pperrors[23];

  // CHECK: dpct::err0 errors1[23];
  hipError_t errors1[23];
  // CHECK: const dpct::err0 *perrors1[23];
  const hipError_t *perrors1[23];
  // CHECK: const dpct::err0 **pperrors1[23];
  const hipError_t **pperrors1[23];

  // CHECK: sycl::range<3> dims[23];
  dim3 dims[23];
  // CHECK: const sycl::range<3> *pdims[23];
  const dim3 *pdims[23];
  // CHECK: const sycl::range<3> **ppdims[23];
  const dim3 **ppdims[23];
};

// CHECK:  void foo(dpct::device_info p) {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: dpct::err0 e;
hipError_t e;

// CHECK: dpct::err0 ee;
hipError_t ee;

// CHECK: dpct::err0 foo_0(dpct::err0);
hipError_t foo_0(hipError_t);

// CHECK: dpct::err0 foo_1(dpct::err0);
hipError_t foo_1(hipError_t);

// CHECK: dpct::err0 apicall(int i) {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: dpct::err0 err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

int main(int argc, char **argv) {
  //CHECK:sycl::range<3> d3(1, 1, 1);
  //CHECK-NEXT:int a = sizeof(sycl::range<3>);
  //CHECK-NEXT:a = sizeof(d3);
  //CHECK-NEXT:a = sizeof d3;
  dim3 d3;
  int a = sizeof(dim3);
  a = sizeof(d3);
  a = sizeof d3;

  //CHECK:dpct::err0 cudaErr_t;
  //CHECK-NEXT:a = sizeof(dpct::err0);
  //CHECK-NEXT:a = sizeof(cudaErr_t);
  //CHECK-NEXT:a = sizeof cudaErr_t;
  hipError_t cudaErr_t;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr_t);
  a = sizeof cudaErr_t;

  //CHECK:int res;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(res);
  //CHECK-NEXT:a = sizeof res;
  hipError_t res;
  a = sizeof(hipError_t);
  a = sizeof(res);
  a = sizeof res;

  //CHECK:int context;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(context);
  //CHECK-NEXT:a = sizeof context;
  hipCtx_t context;
  a = sizeof(hipCtx_t);
  a = sizeof(context);
  a = sizeof context;

  //CHECK:dpct::event_ptr event;
  //CHECK-NEXT:a = sizeof(dpct::event_ptr);
  //CHECK-NEXT:a = sizeof(event);
  //CHECK-NEXT:a = sizeof event;
  hipEvent_t event;
  a = sizeof(hipEvent_t);
  a = sizeof(event);
  a = sizeof event;

  //CHECK:dpct::queue_ptr stream;
  //CHECK-NEXT:a = sizeof(dpct::queue_ptr);
  //CHECK-NEXT:a = sizeof(stream);
  //CHECK-NEXT:a = sizeof stream;
  hipStream_t stream;
  a = sizeof(hipStream_t);
  a = sizeof(stream);
  a = sizeof stream;

  //CHECK:dpct::err0 cudaErr;
  //CHECK-NEXT:a = sizeof(dpct::err0);
  //CHECK-NEXT:a = sizeof(cudaErr);
  //CHECK-NEXT:a = sizeof cudaErr;
  hipError_t cudaErr;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr);
  a = sizeof cudaErr;

  //CHECK:sycl::half h;
  //CHECK-NEXT:a = sizeof(sycl::half);
  //CHECK-NEXT:a = sizeof(h);
  //CHECK-NEXT:a = sizeof h;
  half h;
  a = sizeof(half);
  a = sizeof(h);
  a = sizeof h;

  //CHECK:sycl::half2 h2;
  //CHECK-NEXT:a = sizeof(sycl::half2);
  //CHECK-NEXT:a = sizeof(h2);
  //CHECK-NEXT:a = sizeof h2;
  half2 h2;
  a = sizeof(half2);
  a = sizeof(h2);
  a = sizeof h2;

  //CHECK:int blasStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus);
  //CHECK-NEXT:a = sizeof blasStatus;
  hipblasStatus_t blasStatus;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus);
  a = sizeof blasStatus;

  //CHECK:sycl::float2 complex;
  //CHECK-NEXT:a = sizeof(sycl::float2);
  //CHECK-NEXT:a = sizeof(complex);
  //CHECK-NEXT:a = sizeof complex;
  hipComplex complex;
  a = sizeof(hipComplex);
  a = sizeof(complex);
  a = sizeof complex;

  //CHECK:sycl::double2 doubleComplex;
  //CHECK-NEXT:a = sizeof(sycl::double2);
  //CHECK-NEXT:a = sizeof(doubleComplex);
  //CHECK-NEXT:a = sizeof doubleComplex;
  hipDoubleComplex doubleComplex;
  a = sizeof(hipDoubleComplex);
  a = sizeof(doubleComplex);
  a = sizeof doubleComplex;

  //CHECK:oneapi::mkl::uplo fill;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::uplo);
  //CHECK-NEXT:a = sizeof(fill);
  //CHECK-NEXT:a = sizeof fill;
  hipblasFillMode_t fill;
  a = sizeof(hipblasFillMode_t);
  a = sizeof(fill);
  a = sizeof fill;

  //CHECK:oneapi::mkl::diag diag;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::diag);
  //CHECK-NEXT:a = sizeof(diag);
  //CHECK-NEXT:a = sizeof diag;
  hipblasDiagType_t diag;
  a = sizeof(hipblasDiagType_t);
  a = sizeof(diag);
  a = sizeof diag;

  //CHECK:oneapi::mkl::side side;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::side);
  //CHECK-NEXT:a = sizeof(side);
  //CHECK-NEXT:a = sizeof side;
  hipblasSideMode_t side;
  a = sizeof(hipblasSideMode_t);
  a = sizeof(side);
  a = sizeof side;

  //CHECK:oneapi::mkl::transpose oper;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::transpose);
  //CHECK-NEXT:a = sizeof(oper);
  //CHECK-NEXT:a = sizeof oper;
  hipblasOperation_t oper;
  a = sizeof(hipblasOperation_t);
  a = sizeof(oper);
  a = sizeof oper;

  //CHECK:int blasStatus_legacy;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus_legacy);
  //CHECK-NEXT:a = sizeof blasStatus_legacy;
  hipblasStatus_t blasStatus_legacy;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus_legacy);
  a = sizeof blasStatus_legacy;

  //CHECK:int solverStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(solverStatus);
  //CHECK-NEXT:a = sizeof solverStatus;
  hipsolverStatus_t solverStatus;
  a = sizeof(hipsolverStatus_t);
  a = sizeof(solverStatus);
  a = sizeof solverStatus;

  //CHECK:int64_t eigtype;
  //CHECK-NEXT:a = sizeof(int64_t);
  //CHECK-NEXT:a = sizeof(eigtype);
  //CHECK-NEXT:a = sizeof eigtype;
  hipsolverEigType_t eigtype;
  a = sizeof(hipsolverEigType_t);
  a = sizeof(eigtype);
  a = sizeof eigtype;

  //CHECK:oneapi::mkl::job eigmode;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::job);
  //CHECK-NEXT:a = sizeof(eigmode);
  //CHECK-NEXT:a = sizeof eigmode;
  hipsolverEigMode_t eigmode;
  a = sizeof(hipsolverEigMode_t);
  a = sizeof(eigmode);
  a = sizeof eigmode;

  //CHECK:int randstatus_t;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus_t);
  //CHECK-NEXT:a = sizeof randstatus_t;
  hiprandStatus_t randstatus_t;
  a = sizeof(hiprandStatus_t);
  a = sizeof(randstatus_t);
  a = sizeof randstatus_t;

  //CHECK:dpct::err0 cudaerror;
  //CHECK-NEXT:a = sizeof(dpct::err0);
  //CHECK-NEXT:a = sizeof(cudaerror);
  //CHECK-NEXT:a = sizeof cudaerror;
  hipError_t cudaerror;
  a = sizeof(hipError_t);
  a = sizeof(cudaerror);
  a = sizeof cudaerror;

  //CHECK:int fftresult;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(fftresult);
  //CHECK-NEXT:a = sizeof fftresult;
  hipfftResult_t fftresult;
  a = sizeof(hipfftResult_t);
  a = sizeof(fftresult);
  a = sizeof fftresult;

  //CHECK:hipError_t error_enum;
  //CHECK-NEXT:a = sizeof(hipError_t);
  //CHECK-NEXT:a = sizeof(error_enum);
  //CHECK-NEXT:a = sizeof error_enum;
  hipError_t error_enum;
  a = sizeof(hipError_t);
  a = sizeof(error_enum);
  a = sizeof error_enum;

  //CHECK:int randstatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus);
  //CHECK-NEXT:a = sizeof randstatus;
  hiprandStatus randstatus;
  a = sizeof(hiprandStatus);
  a = sizeof(randstatus);
  a = sizeof randstatus;

  //CHECK:dpct::device_info dp;
  //CHECK-NEXT:a = sizeof(dpct::device_info);
  //CHECK-NEXT:a = sizeof(dp);
  //CHECK-NEXT:a = sizeof dp;
  hipDeviceProp_t dp;
  a = sizeof(hipDeviceProp_t);
  a = sizeof(dp);
  a = sizeof dp;

  //CHECK:sycl::queue *stream_st;
  //CHECK-NEXT:a = sizeof(sycl::queue *);
  //CHECK-NEXT:a = sizeof(stream_st);
  //CHECK-NEXT:a = sizeof stream_st;
  ihipStream_t *stream_st;
  a = sizeof(ihipStream_t*);
  a = sizeof(stream_st);
  a = sizeof stream_st;

  //CHECK:sycl::event *event_st;
  //CHECK-NEXT:a = sizeof(sycl::event *);
  //CHECK-NEXT:a = sizeof(event_st);
  //CHECK-NEXT:a = sizeof event_st;
  ihipEvent_t *event_st;
  a = sizeof(ihipEvent_t*);
  a = sizeof(event_st);
  a = sizeof event_st;

  //CHECK:dpct::blas::descriptor_ptr blashandle;
  //CHECK-NEXT:a = sizeof(dpct::blas::descriptor_ptr);
  //CHECK-NEXT:a = sizeof(blashandle);
  //CHECK-NEXT:a = sizeof blashandle;
  hipblasHandle_t blashandle;
  a = sizeof(hipblasHandle_t);
  a = sizeof(blashandle);
  a = sizeof blashandle;

  //CHECK:dpct::queue_ptr solverdnhandle;
  //CHECK-NEXT:a = sizeof(dpct::queue_ptr);
  //CHECK-NEXT:a = sizeof(solverdnhandle);
  //CHECK-NEXT:a = sizeof solverdnhandle;
  hipsolverHandle_t solverdnhandle;
  a = sizeof(hipsolverHandle_t);
  a = sizeof(solverdnhandle);
  a = sizeof solverdnhandle;

  MY_ERROR_CHECKER(apicall(0));
  return 0;
}

__global__ void foo() {
  void *p;
  // CHECK: (dpct::queue_ptr) p;
  // CHECK-NEXT: (dpct::queue_ptr *)p;
  // CHECK-NEXT: (dpct::queue_ptr **)p;
  // CHECK-NEXT: (dpct::queue_ptr ***)p;
  (hipStream_t)p;
  (hipStream_t *)p;
  (hipStream_t **)p;
  (hipStream_t ***)p;


  // CHECK: malloc(sizeof(dpct::queue_ptr *));
  // CHECK: malloc(sizeof(dpct::queue_ptr **));
  // CHECK: malloc(sizeof(dpct::queue_ptr ***));
  // CHECK: malloc(sizeof(dpct::queue_ptr &));
  malloc(sizeof(hipStream_t *));
  malloc(sizeof(hipStream_t **));
  malloc(sizeof(hipStream_t ***));
  malloc(sizeof(hipStream_t &));

  int i;
  // CHECK: (dpct::err0) i;
  // CHECK-NEXT: (dpct::err0 *)p;
  // CHECK-NEXT: (dpct::err0 **)p;
  // CHECK-NEXT: (dpct::err0 ***)p;
  (hipError_t)i;
  (hipError_t *)p;
  (hipError_t **)p;
  (hipError_t ***)p;

  hipDeviceProp_t cdp;
  // CHECK: dpct::device_info cdp2 = (dpct::device_info)cdp;
  // CHECK-NEXT: (dpct::device_info *)p;
  // CHECK-NEXT: (dpct::device_info **)p;
  // CHECK-NEXT: (dpct::device_info ***)p;
  hipDeviceProp_t cdp2 = (hipDeviceProp_t)cdp;
  (hipDeviceProp_t *)p;
  (hipDeviceProp_t **)p;
  (hipDeviceProp_t ***)p;
}

template <typename T> struct S {};

// CHECK: template <> struct S<dpct::queue_ptr> {};
// CHECK-NEXT: template <> struct S<sycl::queue> {};
// CHECK-NEXT: template <> struct S<sycl::float2> {};
// CHECK-NEXT: template <> struct S<sycl::float4> {};
template <> struct S<hipStream_t> {};
template <> struct S<ihipStream_t> {};
template <> struct S<float2> {};
template <> struct S<float4> {};

void foobar() {
  // CHECK: S<dpct::queue_ptr> s0;
  S<hipStream_t> s0;
  // CHECK: S<sycl::float2> s1;
  S<float2> s1;
  // CHECK: S<sycl::float4> s2;
  S<float4> s2;
}

void fun() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  // CHECK: dpct::queue_ptr *p, s, &r = s;
  hipStream_t *p, s, &r = s;
  // CHECK: dpct::queue_ptr const s_2 = &q_ct1, *p_2, &r_2 = s;
  hipStream_t const s_2 = NULL, *p_2, &r_2 = s;
  // CHECK: const dpct::queue_ptr &r_3 = s, *p_3, s_3 = &q_ct1;
  const hipStream_t &r_3 = s, *p_3, s_3 = NULL;

  // CHECK: dpct::queue_ptr const *pc, sc = s, &rc = s;
  hipStream_t const *pc, sc = s, &rc = s;
  // CHECK: const dpct::queue_ptr *pc1, sc1 = s, &rc1 = s;
  const hipStream_t *pc1, sc1 = s, &rc1 = s;
  // CHECK: dpct::queue_ptr s1, *p1, &r1 = *p1;
  hipStream_t s1, *p1, &r1 = *p1;
  // CHECK: dpct::queue_ptr &r2 = s1, *p2, s2;
  hipStream_t &r2 = s1, *p2, s2;

  // CHECK: dpct::queue_ptr &r3 = s2,
  // CHECK-NEXT:             *p3,
  // CHECK-NEXT:             s3;
  hipStream_t &r3 = s2,
               *p3,
               s3;

  // CHECK: dpct::queue_ptr const s4 = s1, s5 = s2;
  hipStream_t const s4 = s1, s5 = s2;
  // CHECK: const dpct::queue_ptr s6 = s1, s7 = s2;
  const hipStream_t s6 = s1, s7 = s2;

  // CHECK: dpct::queue_ptr const *s8, *s9;
  hipStream_t const *s8, *s9;
  // CHECK: const dpct::queue_ptr *s10, *s11;
  const hipStream_t *s10, *s11;
  // CHECK: dpct::queue_ptr *const s12 = NULL, *const s13 = NULL;
  hipStream_t *const s12 = NULL, *const s13 = NULL;
  // CHECK: const dpct::queue_ptr *const s14 = NULL, *const s15 = NULL;
  const hipStream_t *const s14 = NULL, *const s15 = NULL;
}

void fun2() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  // CHECK: dpct::queue_ptr s, s2;
  hipStream_t s, s2;
  // CHECK: dpct::queue_ptr const s3 = &q_ct1, s4 = &q_ct1;
  hipStream_t const s3 = NULL, s4 = NULL;
  // CHECK: const dpct::queue_ptr s5 = &q_ct1, s6 = &q_ct1;
  const hipStream_t s5 = NULL, s6 = NULL;

  // CHECK: dpct::queue_ptr *s7, *const s8 = NULL;
  hipStream_t *s7, *const s8 = NULL;
  // CHECK: dpct::queue_ptr *const s9 = NULL, *s10;
  hipStream_t *const s9 = NULL, *s10;
  // CHECK: const dpct::queue_ptr *s11, *const s12 = NULL;
  const hipStream_t *s11, *const s12 = NULL;
  // CHECK: dpct::queue_ptr const *const s13 = NULL, *s14;
  hipStream_t const *const s13 = NULL, *s14;
  // CHECK: const dpct::queue_ptr *const s15 = NULL, *s16;
  const hipStream_t *const s15 = NULL, *s16;
  // CHECK: dpct::queue_ptr const *s17, *const s18 = NULL;
  hipStream_t const *s17, *const s18 = NULL;
}

// CHECK:template <>
// CHECK-NEXT:struct S<int &&> {};
// CHECK-NEXT:template <> struct S<dpct::err0> {};
// CHECK-NEXT:template <> struct S<dpct::err0 *> {};
// CHECK-NEXT:template <> struct S<dpct::err0 &> {};
// CHECK-NEXT:template <> struct S<dpct::err0 &&> {};
template <>
struct S<int &&> {};
template <> struct S<hipError_t> {};
template <> struct S<hipError_t *> {};
template <> struct S<hipError_t &> {};
template <> struct S<hipError_t &&> {};

// CHECK: template <int SMEM_CONFIG = 0>
// CHECK-NEXT: class BlockRadixRank0 {};
// CHECK-NEXT: template <int SMEM_CONFIG = 1>
// CHECK-NEXT: class BlockRadixRank1 {};
// CHECK-NEXT: template <int SMEM_CONFIG = 2>
// CHECK-NEXT: class BlockRadixRank2 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeDefault>
class BlockRadixRank0 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeFourByte>
class BlockRadixRank1 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeEightByte>
class BlockRadixRank2 {};


void fun3() {
  char devstr[128] = "";
  // CHECK: dpct::device_info deviceProp;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support a device property functionally compatible with pciDomainID. It was migrated to -1. You may need to adjust the value of -1 for the specific device.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support a device property functionally compatible with pciBusID. It was migrated to -1. You may need to adjust the value of -1 for the specific device.
  // CHECK-NEXT: */
  // CHECK-NEXT: sprintf(devstr, "pci %x:%x:%x", -1, -1, deviceProp.get_device_id());
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support a device property functionally compatible with concurrentKernels. It was migrated to true. You may need to adjust the value of true for the specific device.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (true) {
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support a device property functionally compatible with canMapHostMemory. It was migrated to false. You may need to adjust the value of false for the specific device.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (!false) {
  // CHECK-NEXT: }
  hipDeviceProp_t deviceProp;
  sprintf(devstr, "pci %x:%x:%x", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);
  if (deviceProp.concurrentKernels) {
  }
  if (!deviceProp.canMapHostMemory){
  }
}

void fun4() {
  // CHECK: std::vector<dpct::queue_ptr> vec1;
  // CHECK-NEXT: vec1.push_back(dpct::queue_ptr());
  // CHECK-NEXT: std::vector<dpct::queue_ptr> vec2;
  // CHECK-NEXT: vec2.push_back(dpct::queue_ptr());
  // CHECK-NEXT: dpct::queue_ptr a1 = dpct::queue_ptr();
  // CHECK-NEXT: dpct::queue_ptr a2 = dpct::queue_ptr();
  std::vector<hipStream_t> vec1;
  vec1.push_back(hipStream_t());
  std::vector<hipStream_t> vec2;
  vec2.push_back(hipStream_t());
  hipStream_t a1 = hipStream_t();
  hipStream_t a2 = hipStream_t();
}

namespace {
// CHECK: dpct::memcpy_direction K;
// CHECK-NEXT: dpct::memcpy_direction fun(dpct::memcpy_direction);
hipMemcpyKind K;
hipMemcpyKind fun(hipMemcpyKind);
}


namespace {
// CHECK: int M;
// CHECK-NEXT: int fun(int);
hipComputeMode M;
hipComputeMode fun(hipComputeMode);
}

// CHECK: void foo_2(dpct::library_data_t a1, dpct::library_data_t a2, dpct::library_data_t a3) {
// CHECK-NEXT:   dpct::library_data_t b1 = a1;
// CHECK-NEXT:   dpct::library_data_t b2 = a2;
// CHECK-NEXT:   dpct::library_data_t b3 = a3;
// CHECK-NEXT: }
void foo_2(hipDataType a1, hipDataType a2, hipDataType a3) {
  hipDataType b1 = a1;
  hipDataType b2 = a2;
  hipDataType b3 = a3;
}

__device__ void foo_3() {
  // CHECK: sycl::range<3> d3 = {3, 2, 1}, *pd3 = &d3;
  dim3 d3 = {1, 2, 3}, *pd3 = &d3;
  int64_t m = 0;
  // CHECK: m = std::min(m, int64_t((*pd3)[2]));
  // CHECK-NEXT: m = std::min(m, int64_t((*pd3)[1]));
  // CHECK-NEXT: m = std::min(m, int64_t((*pd3)[0]));
  // CHECK-NEXT: m = std::min(m, int64_t(d3[2]));
  // CHECK-NEXT: m = std::min(m, int64_t(d3[1]));
  // CHECK-NEXT: m = std::min(m, int64_t(d3[0]));
  m = std::min(m, int64_t{pd3->x});
  m = std::min(m, int64_t{pd3->y});
  m = std::min(m, int64_t{pd3->z});
  m = std::min(m, int64_t{d3.x});
  m = std::min(m, int64_t{d3.y});
  m = std::min(m, int64_t{d3.z});
}

template <typename integer>
constexpr inline integer ceil_div(integer n, integer m) {
  return (n + m - 1) / m;
}

void foo_4() {
  const int64_t num_irows = 32;
  const int64_t num_orows = 32;
  // CHECK: sycl::range<3> threads(1, 1, 32);
  dim3 threads(32);
  int64_t maxGridDim = 1024;
  // CHECK: sycl::range<3> grid_1(1, std::min(maxGridDim, ceil_div(num_irows, int64_t(threads[2]))), std::min(maxGridDim, num_orows));
  dim3 grid_1(std::min(maxGridDim, num_orows), std::min(maxGridDim, ceil_div(num_irows, int64_t{threads.x})));

  int row_size = 16;
  // CHECK: sycl::range<3> grid_2(1, 1, std::min<int>(maxGridDim, ceil_div(row_size, int(threads[1]))));
  dim3 grid_2(std::min<int>(maxGridDim, ceil_div(row_size, int(threads.y))));

  // CHECK: int64_t m = int64_t(threads[1]);
  int64_t m = int64_t{threads.y};
  // CHECK: m = int64_t(threads[1]);
  m = int64_t{threads.y};
  typedef int64_t MY_INT64;
  // CHECK: m = std::min(int64_t(threads[2]), MY_INT64(threads[0]));
  m = std::min(int64_t{threads.x}, MY_INT64{threads.z});

  int num = 1024;
  // CHECK: m = int64_t{num};
  m = int64_t{num};
  // CHECK: m = std::min(int64_t(threads[2]), MY_INT64{num});
  m = std::min(int64_t{threads.x}, MY_INT64{num});

  struct CFoo {
    int64_t a = 0;
    CFoo(int64_t b) : a(b) {}
    operator int64_t() { return a; }
  };
  // CHECK: CFoo cfoo{num};
  CFoo cfoo{num};
  // CHECK: m = std::min(int64_t(threads[2]), int64_t{cfoo});
  m = std::min(int64_t{threads.x}, int64_t{cfoo});
}


// CHECK: dpct::device_ptr a = (dpct::device_ptr)0, b = (dpct::device_ptr)0;
hipDeviceptr_t a = (hipDeviceptr_t)0, b = (hipDeviceptr_t)0;
void test() {
  // CHECK: dpct::device_ptr a = (dpct::device_ptr)0, b = (dpct::device_ptr)0;
  hipDeviceptr_t a = (hipDeviceptr_t)0, b = (hipDeviceptr_t)0;
}
