#include "hip/hip_runtime.h"
// RUN: dpct -out-root %T/types001 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -fno-delayed-template-parsing
// RUN: FileCheck %s --match-full-lines --input-file %T/types001/types001.dp.cpp

#include <hip/hip_runtime.h>
#include <hipblas.h>
#include <hipblas.h>
#include <hiprand.h>
#include <hipsolver.h>
#include <hipfft/hipfft.h>
#include <stdio.h>
#include <vector>

// CHECK: dpct::device_info deviceProp;
hipDeviceProp_t deviceProp;

// CHECK: const dpct::device_info deviceProp1 = {};
const hipDeviceProp_t deviceProp1 = {};

// CHECK: volatile dpct::device_info deviceProp2;
volatile hipDeviceProp_t deviceProp2;

// CHDCK: sycl::event events[23];
hipEvent_t events[23];
// CHECK: const sycl::event *pevents[23];
const hipEvent_t *pevents[23];
// CHECK: const sycl::event **ppevents[23];
const hipEvent_t **ppevents[23];

// CHECK: int errors[23];
hipError_t errors[23];
// CHECK: const int *perrors[23];
const hipError_t *perrors[23];
// CHECK: const int **pperrors[23];
const hipError_t **pperrors[23];

// CHECK: int errors1[23];
hipError_t errors1[23];
// CHECK: const int *perrors1[23];
const hipError_t *perrors1[23];
// CHECK: const int **pperrors1[23];
const hipError_t **pperrors1[23];

// CHECK: sycl::range<3> dims[23];
dim3 dims[23];
// CHECK: const sycl::range<3> *pdims[23];
const dim3 *pdims[23];
// CHECK: const sycl::range<3> **ppdims[23];
const dim3 **ppdims[23];

struct s {
  // CHECK: sycl::event events[23];
  hipEvent_t events[23];
  // CHECK: const sycl::event *pevents[23];
  const hipEvent_t *pevents[23];
  // CHECK: const sycl::event **ppevents[23];
  const hipEvent_t **ppevents[23];

  // CHECK: int errors[23];
  hipError_t errors[23];
  // CHECK: const int *perrors[23];
  const hipError_t *perrors[23];
  // CHECK: const int **pperrors[23];
  const hipError_t **pperrors[23];

  // CHECK: int errors1[23];
  hipError_t errors1[23];
  // CHECK: const int *perrors1[23];
  const hipError_t *perrors1[23];
  // CHECK: const int **pperrors1[23];
  const hipError_t **pperrors1[23];

  // CHECK: sycl::range<3> dims[23];
  dim3 dims[23];
  // CHECK: const sycl::range<3> *pdims[23];
  const dim3 *pdims[23];
  // CHECK: const sycl::range<3> **ppdims[23];
  const dim3 **ppdims[23];
};

// CHECK:  void foo(dpct::device_info p) {
void foo(hipDeviceProp_t p) {
  return;
}

// CHECK: int e;
hipError_t e;

// CHECK: int ee;
hipError_t ee;

// CHECK: int foo_0(int);
hipError_t foo_0(hipError_t);

// CHECK: int foo_1(int);
hipError_t foo_1(hipError_t);

// CHECK: int apicall(int i) {
hipError_t apicall(int i) {
  return hipSuccess;
};

// CHECK: int err = apicall(0);
hipError_t err = apicall(0);

template <typename T>
// CHECK: void my_error_checker(T ReturnValue, char const *const FuncName) {
void my_error_checker(T ReturnValue, char const *const FuncName) {
}

#define MY_ERROR_CHECKER(CALL) my_error_checker((CALL), #CALL)

int main(int argc, char **argv) {
  //CHECK:sycl::range<3> d3(1, 1, 1);
  //CHECK-NEXT:int a = sizeof(sycl::range<3>);
  //CHECK-NEXT:a = sizeof(d3);
  //CHECK-NEXT:a = sizeof d3;
  dim3 d3;
  int a = sizeof(dim3);
  a = sizeof(d3);
  a = sizeof d3;

  //CHECK:int cudaErr_t;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaErr_t);
  //CHECK-NEXT:a = sizeof cudaErr_t;
  hipError_t cudaErr_t;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr_t);
  a = sizeof cudaErr_t;

  //CHECK:int res;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(res);
  //CHECK-NEXT:a = sizeof res;
  hipError_t res;
  a = sizeof(hipError_t);
  a = sizeof(res);
  a = sizeof res;

  //CHECK:int context;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(context);
  //CHECK-NEXT:a = sizeof context;
  hipCtx_t context;
  a = sizeof(hipCtx_t);
  a = sizeof(context);
  a = sizeof context;

  //CHECK:sycl::event event;
  //CHECK-NEXT:a = sizeof(sycl::event);
  //CHECK-NEXT:a = sizeof(event);
  //CHECK-NEXT:a = sizeof event;
  hipEvent_t event;
  a = sizeof(hipEvent_t);
  a = sizeof(event);
  a = sizeof event;

  //CHECK:sycl::queue *stream;
  //CHECK-NEXT:a = sizeof(sycl::queue *);
  //CHECK-NEXT:a = sizeof(stream);
  //CHECK-NEXT:a = sizeof stream;
  hipStream_t stream;
  a = sizeof(hipStream_t);
  a = sizeof(stream);
  a = sizeof stream;

  //CHECK:int cudaErr;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaErr);
  //CHECK-NEXT:a = sizeof cudaErr;
  hipError_t cudaErr;
  a = sizeof(hipError_t);
  a = sizeof(cudaErr);
  a = sizeof cudaErr;

  //CHECK:sycl::half h;
  //CHECK-NEXT:a = sizeof(sycl::half);
  //CHECK-NEXT:a = sizeof(h);
  //CHECK-NEXT:a = sizeof h;
  half h;
  a = sizeof(half);
  a = sizeof(h);
  a = sizeof h;

  //CHECK:sycl::half2 h2;
  //CHECK-NEXT:a = sizeof(sycl::half2);
  //CHECK-NEXT:a = sizeof(h2);
  //CHECK-NEXT:a = sizeof h2;
  half2 h2;
  a = sizeof(half2);
  a = sizeof(h2);
  a = sizeof h2;

  //CHECK:int blasStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus);
  //CHECK-NEXT:a = sizeof blasStatus;
  hipblasStatus_t blasStatus;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus);
  a = sizeof blasStatus;

  //CHECK:sycl::float2 complex;
  //CHECK-NEXT:a = sizeof(sycl::float2);
  //CHECK-NEXT:a = sizeof(complex);
  //CHECK-NEXT:a = sizeof complex;
  hipComplex complex;
  a = sizeof(hipComplex);
  a = sizeof(complex);
  a = sizeof complex;

  //CHECK:sycl::double2 doubleComplex;
  //CHECK-NEXT:a = sizeof(sycl::double2);
  //CHECK-NEXT:a = sizeof(doubleComplex);
  //CHECK-NEXT:a = sizeof doubleComplex;
  hipDoubleComplex doubleComplex;
  a = sizeof(hipDoubleComplex);
  a = sizeof(doubleComplex);
  a = sizeof doubleComplex;

  //CHECK:oneapi::mkl::uplo fill;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::uplo);
  //CHECK-NEXT:a = sizeof(fill);
  //CHECK-NEXT:a = sizeof fill;
  hipblasFillMode_t fill;
  a = sizeof(hipblasFillMode_t);
  a = sizeof(fill);
  a = sizeof fill;

  //CHECK:oneapi::mkl::diag diag;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::diag);
  //CHECK-NEXT:a = sizeof(diag);
  //CHECK-NEXT:a = sizeof diag;
  hipblasDiagType_t diag;
  a = sizeof(hipblasDiagType_t);
  a = sizeof(diag);
  a = sizeof diag;

  //CHECK:oneapi::mkl::side side;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::side);
  //CHECK-NEXT:a = sizeof(side);
  //CHECK-NEXT:a = sizeof side;
  hipblasSideMode_t side;
  a = sizeof(hipblasSideMode_t);
  a = sizeof(side);
  a = sizeof side;

  //CHECK:oneapi::mkl::transpose oper;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::transpose);
  //CHECK-NEXT:a = sizeof(oper);
  //CHECK-NEXT:a = sizeof oper;
  hipblasOperation_t oper;
  a = sizeof(hipblasOperation_t);
  a = sizeof(oper);
  a = sizeof oper;

  //CHECK:int blasStatus_legacy;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(blasStatus_legacy);
  //CHECK-NEXT:a = sizeof blasStatus_legacy;
  hipblasStatus_t blasStatus_legacy;
  a = sizeof(hipblasStatus_t);
  a = sizeof(blasStatus_legacy);
  a = sizeof blasStatus_legacy;

  //CHECK:int solverStatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(solverStatus);
  //CHECK-NEXT:a = sizeof solverStatus;
  hipsolverStatus_t solverStatus;
  a = sizeof(hipsolverStatus_t);
  a = sizeof(solverStatus);
  a = sizeof solverStatus;

  //CHECK:int64_t eigtype;
  //CHECK-NEXT:a = sizeof(int64_t);
  //CHECK-NEXT:a = sizeof(eigtype);
  //CHECK-NEXT:a = sizeof eigtype;
  hipsolverEigType_t eigtype;
  a = sizeof(hipsolverEigType_t);
  a = sizeof(eigtype);
  a = sizeof eigtype;

  //CHECK:oneapi::mkl::job eigmode;
  //CHECK-NEXT:a = sizeof(oneapi::mkl::job);
  //CHECK-NEXT:a = sizeof(eigmode);
  //CHECK-NEXT:a = sizeof eigmode;
  hipsolverEigMode_t eigmode;
  a = sizeof(hipsolverEigMode_t);
  a = sizeof(eigmode);
  a = sizeof eigmode;

  //CHECK:int randstatus_t;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus_t);
  //CHECK-NEXT:a = sizeof randstatus_t;
  hiprandStatus_t randstatus_t;
  a = sizeof(hiprandStatus_t);
  a = sizeof(randstatus_t);
  a = sizeof randstatus_t;

  //CHECK:int cudaerror;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(cudaerror);
  //CHECK-NEXT:a = sizeof cudaerror;
  hipError_t cudaerror;
  a = sizeof(hipError_t);
  a = sizeof(cudaerror);
  a = sizeof cudaerror;

  //CHECK:int fftresult;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(fftresult);
  //CHECK-NEXT:a = sizeof fftresult;
  hipfftResult_t fftresult;
  a = sizeof(hipfftResult_t);
  a = sizeof(fftresult);
  a = sizeof fftresult;

  //CHECK:hipError_t error_enum;
  //CHECK-NEXT:a = sizeof(hipError_t);
  //CHECK-NEXT:a = sizeof(error_enum);
  //CHECK-NEXT:a = sizeof error_enum;
  hipError_t error_enum;
  a = sizeof(hipError_t);
  a = sizeof(error_enum);
  a = sizeof error_enum;

  //CHECK:int randstatus;
  //CHECK-NEXT:a = sizeof(int);
  //CHECK-NEXT:a = sizeof(randstatus);
  //CHECK-NEXT:a = sizeof randstatus;
  hiprandStatus randstatus;
  a = sizeof(hiprandStatus);
  a = sizeof(randstatus);
  a = sizeof randstatus;

  //CHECK:dpct::device_info dp;
  //CHECK-NEXT:a = sizeof(dpct::device_info);
  //CHECK-NEXT:a = sizeof(dp);
  //CHECK-NEXT:a = sizeof dp;
  hipDeviceProp_t dp;
  a = sizeof(hipDeviceProp_t);
  a = sizeof(dp);
  a = sizeof dp;

  //CHECK:sycl::queue *stream_st;
  //CHECK-NEXT:a = sizeof(sycl::queue *);
  //CHECK-NEXT:a = sizeof(stream_st);
  //CHECK-NEXT:a = sizeof stream_st;
  ihipStream_t *stream_st;
  a = sizeof(ihipStream_t*);
  a = sizeof(stream_st);
  a = sizeof stream_st;

  //CHECK:sycl::event *event_st;
  //CHECK-NEXT:a = sizeof(sycl::event *);
  //CHECK-NEXT:a = sizeof(event_st);
  //CHECK-NEXT:a = sizeof event_st;
  ihipEvent_t *event_st;
  a = sizeof(ihipEvent_t*);
  a = sizeof(event_st);
  a = sizeof event_st;

  //CHECK:sycl::queue *blashandle;
  //CHECK-NEXT:a = sizeof(sycl::queue *);
  //CHECK-NEXT:a = sizeof(blashandle);
  //CHECK-NEXT:a = sizeof blashandle;
  hipblasHandle_t blashandle;
  a = sizeof(hipblasHandle_t);
  a = sizeof(blashandle);
  a = sizeof blashandle;

  //CHECK:sycl::queue *solverdnhandle;
  //CHECK-NEXT:a = sizeof(sycl::queue *);
  //CHECK-NEXT:a = sizeof(solverdnhandle);
  //CHECK-NEXT:a = sizeof solverdnhandle;
  hipsolverHandle_t solverdnhandle;
  a = sizeof(hipsolverHandle_t);
  a = sizeof(solverdnhandle);
  a = sizeof solverdnhandle;

  MY_ERROR_CHECKER(apicall(0));
  return 0;
}

__global__ void foo() {
  void *p;
  // CHECK: (sycl::queue *)p;
  // CHECK-NEXT: (sycl::queue **)p;
  // CHECK-NEXT: (sycl::queue ***)p;
  // CHECK-NEXT: (sycl::queue ****)p;
  (hipStream_t)p;
  (hipStream_t *)p;
  (hipStream_t **)p;
  (hipStream_t ***)p;


  // CHECK: malloc(sizeof(sycl::queue **));
  // CHECK-NEXT: malloc(sizeof(sycl::queue ***));
  // CHECK-NEXT: malloc(sizeof(sycl::queue ****));
  // CHECK-NEXT: malloc(sizeof(sycl::queue *&));
  malloc(sizeof(hipStream_t *));
  malloc(sizeof(hipStream_t **));
  malloc(sizeof(hipStream_t ***));
  malloc(sizeof(hipStream_t &));

  int i;
  // CHECK: (int)i;
  // CHECK-NEXT: (int *)p;
  // CHECK-NEXT: (int **)p;
  // CHECK-NEXT: (int ***)p;
  (hipError_t)i;
  (hipError_t *)p;
  (hipError_t **)p;
  (hipError_t ***)p;

  hipDeviceProp_t cdp;
  // CHECK: dpct::device_info cdp2 = (dpct::device_info)cdp;
  // CHECK-NEXT: (dpct::device_info *)p;
  // CHECK-NEXT: (dpct::device_info **)p;
  // CHECK-NEXT: (dpct::device_info ***)p;
  hipDeviceProp_t cdp2 = (hipDeviceProp_t)cdp;
  (hipDeviceProp_t *)p;
  (hipDeviceProp_t **)p;
  (hipDeviceProp_t ***)p;
}

template <typename T> struct S {};

// CHECK: template <> struct S<sycl::queue *> {};
// CHECK-NEXT: template <> struct S<sycl::queue> {};
// CHECK-NEXT: template <> struct S<sycl::float2> {};
// CHECK-NEXT: template <> struct S<sycl::float4> {};
template <> struct S<hipStream_t> {};
template <> struct S<ihipStream_t> {};
template <> struct S<float2> {};
template <> struct S<float4> {};

void foobar() {
  // CHECK: S<sycl::queue *> s0;
  S<hipStream_t> s0;
  // CHECK: S<sycl::float2> s1;
  S<float2> s1;
  // CHECK: S<sycl::float4> s2;
  S<float4> s2;
}

void fun() {
  // CHECK: sycl::queue **p, *s, *&r = s;
  hipStream_t *p, s, &r = s;
  // CHECK: sycl::queue *const s_2 = NULL, *const * p_2, *const &r_2 = s;
  hipStream_t const s_2 = NULL, *p_2, &r_2 = s;
  // CHECK: sycl::queue *const &r_3 = s, *const * p_3, *const s_3 = NULL;
  const hipStream_t &r_3 = s, *p_3, s_3 = NULL;

  // CHECK: sycl::queue *const *pc, *const sc = s, *const &rc = s;
  hipStream_t const *pc, sc = s, &rc = s;
  // CHECK: sycl::queue *const *pc1, *const sc1 = s, *const &rc1 = s;
  const hipStream_t *pc1, sc1 = s, &rc1 = s;
  // CHECK: sycl::queue *s1, **p1, *&r1 = *p1;
  hipStream_t s1, *p1, &r1 = *p1;
  // CHECK: sycl::queue *&r2 = s1, **p2, *s2;
  hipStream_t &r2 = s1, *p2, s2;

  // CHECK: sycl::queue *&r3 = s2,
  // CHECK-NEXT:             **p3,
  // CHECK-NEXT:             *s3;
  hipStream_t &r3 = s2,
               *p3,
               s3;

  // CHECK: sycl::queue *const s4 = s1, *const s5 = s2;
  hipStream_t const s4 = s1, s5 = s2;
  // CHECK: sycl::queue *const s6 = s1, *const s7 = s2;
  const hipStream_t s6 = s1, s7 = s2;

  // CHECK: sycl::queue *const *s8, *const *s9;
  hipStream_t const *s8, *s9;
  // CHECK: sycl::queue *const *s10, *const *s11;
  const hipStream_t *s10, *s11;
  // CHECK: sycl::queue **const s12 = NULL, **const s13 = NULL;
  hipStream_t *const s12 = NULL, *const s13 = NULL;
  // CHECK: sycl::queue *const *const s14 = NULL, *const *const s15 = NULL;
  const hipStream_t *const s14 = NULL, *const s15 = NULL;
}

void fun2() {
  // CHECK: sycl::queue *s, *s2;
  hipStream_t s, s2;
  // CHECK: sycl::queue *const s3 = NULL, *const s4 = NULL;
  hipStream_t const s3 = NULL, s4 = NULL;
  // CHECK: sycl::queue *const s5 = NULL, *const s6 = NULL;
  const hipStream_t s5 = NULL, s6 = NULL;

  // CHECK: sycl::queue **s7, **const s8 = NULL;
  hipStream_t *s7, *const s8 = NULL;
  // CHECK: sycl::queue **const s9 = NULL, **s10;
  hipStream_t *const s9 = NULL, *s10;
  // CHECK: sycl::queue *const *s11, *const *const s12 = NULL;
  const hipStream_t *s11, *const s12 = NULL;
  // CHECK: sycl::queue *const *const s13 = NULL, *const * s14;
  hipStream_t const *const s13 = NULL, *s14;
  // CHECK: sycl::queue *const *const s15 = NULL, *const * s16;
  const hipStream_t *const s15 = NULL, *s16;
  // CHECK: sycl::queue *const *s17, *const *const s18 = NULL;
  hipStream_t const *s17, *const s18 = NULL;
}

// CHECK:template <>
// CHECK-NEXT:struct S<int &&> {};
// CHECK-NEXT:template <> struct S<int> {};
// CHECK-NEXT:template <> struct S<int *> {};
// CHECK-NEXT:template <> struct S<int &> {};
// CHECK-NEXT:template <> struct S<int &&> {};
template <>
struct S<int &&> {};
template <> struct S<hipError_t> {};
template <> struct S<hipError_t *> {};
template <> struct S<hipError_t &> {};
template <> struct S<hipError_t &&> {};

// CHECK: template <int SMEM_CONFIG = 0>
// CHECK-NEXT: class BlockRadixRank0 {};
// CHECK-NEXT: template <int SMEM_CONFIG = 1>
// CHECK-NEXT: class BlockRadixRank1 {};
// CHECK-NEXT: template <int SMEM_CONFIG = 2>
// CHECK-NEXT: class BlockRadixRank2 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeDefault>
class BlockRadixRank0 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeFourByte>
class BlockRadixRank1 {};
template <hipSharedMemConfig SMEM_CONFIG = hipSharedMemBankSizeEightByte>
class BlockRadixRank2 {};


void fun3() {
  char devstr[128] = "";
  // CHECK: dpct::device_info deviceProp;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with pciDomainID. It was migrated to -1. You may need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with pciBusID. It was migrated to -1. You may need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with pciDeviceID. It was migrated to -1. You may need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: sprintf(devstr, "pci %x:%x:%x", -1, -1, -1);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with concurrentKernels. It was migrated to true. You may need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (true) {
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1051:{{[0-9]+}}: SYCL does not support the device property that would be functionally compatible with canMapHostMemory. It was migrated to false. You may need to rewrite the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if (!false) {
  // CHECK-NEXT: }
  hipDeviceProp_t deviceProp;
  sprintf(devstr, "pci %x:%x:%x", deviceProp.pciDomainID, deviceProp.pciBusID, deviceProp.pciDeviceID);
  if (deviceProp.concurrentKernels) {
  }
  if (!deviceProp.canMapHostMemory){
  }
}

void fun4() {
  // CHECK: std::vector<sycl::queue *> vec1;
  // CHECK-NEXT: vec1.push_back(nullptr);
  // CHECK-NEXT: std::vector<sycl::queue *> vec2;
  // CHECK-NEXT: vec2.push_back(nullptr);
  // CHECK-NEXT: sycl::queue *a1 = nullptr;
  // CHECK-NEXT: sycl::queue *a2 = nullptr;
  std::vector<hipStream_t> vec1;
  vec1.push_back(hipStream_t());
  std::vector<hipStream_t> vec2;
  vec2.push_back(hipStream_t());
  hipStream_t a1 = hipStream_t();
  hipStream_t a2 = hipStream_t();
}

namespace {
// CHECK: dpct::memcpy_direction K;
// CHECK-NEXT: dpct::memcpy_direction fun(dpct::memcpy_direction);
hipMemcpyKind K;
hipMemcpyKind fun(hipMemcpyKind);
}


namespace {
// CHECK: int M;
// CHECK-NEXT: int fun(int);
hipComputeMode M;
hipComputeMode fun(hipComputeMode);
}

// CHECK: void foo_2(dpct::library_data_t a1, dpct::library_data_t a2, dpct::library_data_t a3, dpct::library_data_t a4) {
// CHECK-NEXT:   dpct::library_data_t b1 = a1;
// CHECK-NEXT:   dpct::library_data_t b2 = a2;
// CHECK-NEXT:   dpct::library_data_t b3 = a3;
// CHECK-NEXT:   dpct::library_data_t b4 = a4;
// CHECK-NEXT: }
void foo_2(hipDataType a1, hipDataType a2, hipDataType a3, hipblasComputeType_t a4) {
  hipDataType b1 = a1;
  hipDataType b2 = a2;
  hipDataType b3 = a3;
  hipblasComputeType_t b4 = a4;
}
