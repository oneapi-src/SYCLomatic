// FIXME
// RUN: dpct --usm-level=none -out-root %T/ctad %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda --format-range=none --enable-ctad -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/ctad/ctad.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/ctad/ctad.dp.cpp -o %T/ctad/ctad.dp.o %}


#include <hip/hip_runtime.h>
#include <cstdio>
#include <algorithm>

#define NUM 23

// CHECK: void func(dpct::dim3 a, dpct::dim3 b, dpct::dim3 c, dpct::dim3 d) {
void func(dim3 a, dim3 b, dim3 c, dim3 d) {
}

__global__ void kernel(int dim) {
  __shared__ int k[32];
}

int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.out_of_order_queue();
  // range default constructor does the right thing.
  // CHECK: dpct::dim3 deflt;
  dim3 deflt;

  // CHECK:  sycl::range deflt_1{0, 0, 0};
  // CHECK-NEXT: sycl::id deflt_2{0, 0, 0};
  hipExtent deflt_1;
  hipPos deflt_2;

  // CHECK: dpct::dim3 round1_1(NUM);
  dim3 round1_1(NUM);

  hipExtent exten = make_hipExtent(1,1,1);;

  // CHECK: dpct::dim3 castini = (dpct::dim3)4;
  dim3 castini = (dim3)4;

  // CHECK:   sycl::range castini_1 = exten;
  // CHECK-NEXT:sycl::id castini_2 = deflt_2;
  hipExtent castini_1 = exten;
  hipPos castini_2 = deflt_2;

  // CHECK: dpct::dim3 copyctor1 = dpct::dim3((dpct::dim3)33);
  dim3 copyctor1 = dim3((dim3)33);


  // CHECK: dpct::dim3 copyctor2 = dpct::dim3(copyctor1);
  dim3 copyctor2 = dim3(copyctor1);

  // CHECK: dpct::dim3 copyctor3(copyctor1);
  dim3 copyctor3(copyctor1);

  // CHECK: sycl::range copyctor31(exten);
  // CHECK-NEXT: sycl::id copyctor32(deflt_2);
  hipExtent copyctor31(exten);
  hipPos copyctor32(deflt_2);

  // CHECK: func((dpct::dim3)1, dpct::dim3(1), dpct::dim3(2, 1), dpct::dim3(3, 2, 1));
  func((dim3)1, dim3(1), dim3(2, 1), dim3(3, 2, 1));
  // CHECK: func(deflt, dpct::dim3(deflt), (dpct::dim3)deflt, 2 + 3 * 3);
  func(deflt, dim3(deflt), (dim3)deflt, 2 + 3 * 3);

  // CHECK: sycl::range<3> *p_extent = nullptr;
  hipExtent *p_extent = nullptr;

  // CHECK: dpct::dim3 *p = &deflt;
  dim3 *p = &deflt;
  // CHECK: dpct::dim3 **pp = &p;
  dim3 **pp = &p;

  // CHECK: sycl::range<3> *p_1 = &deflt_1;
  // CHECK-NEXT: sycl::id<3> *p_2 = &deflt_2;
  hipExtent *p_1 = &deflt_1;
  hipPos *p_2 = &deflt_2;

  struct  container
  {
    unsigned int x, y, z;
    // CHECK: dpct::dim3 w;
    dim3 w;
    // CHECK: dpct::dim3 *pw;
    dim3 *pw;
    // CHECK: dpct::dim3 **ppw;
    dim3 **ppw;
  };

  // CHECK: dpct::dim3 gpu_blocks(1 / (castini.x * 200));
  dim3 gpu_blocks(1 / (castini.x * 200));
  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::local_accessor<int, 1> k_acc_ct1(sycl::range(32), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range(1, 1, 1), sycl::range(1, 1, 1)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel(1, k_acc_ct1.get_multi_ptr<sycl::access::decorated::no>().get());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel<<<1, 1>>>(1);
  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::local_accessor<int, 1> k_acc_ct1(sycl::range(32), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(sycl::range(1, 1, NUM) * sycl::range(1, 1, NUM), sycl::range(1, 1, NUM)),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel(1, k_acc_ct1.get_multi_ptr<sycl::access::decorated::no>().get());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel<<<NUM, NUM>>>(1);

  // CHECK:   q_ct1.submit(
  // CHECK-NEXT:     [&](sycl::handler &cgh) {
  // CHECK-NEXT:       sycl::local_accessor<int, 1> k_acc_ct1(sycl::range(32), cgh);
  // CHECK-EMPTY:
  // CHECK-NEXT:       cgh.parallel_for<dpct_kernel_name<class kernel_{{[a-f0-9]+}}>>(
  // CHECK-NEXT:         sycl::nd_range<3>(deflt * deflt, deflt),
  // CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:           kernel(1, k_acc_ct1.get_multi_ptr<sycl::access::decorated::no>().get());
  // CHECK-NEXT:         });
  // CHECK-NEXT:     });
  kernel<<<deflt, deflt>>>(1);
}


