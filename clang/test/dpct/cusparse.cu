// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cusparse %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cusparse/cusparse.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
float alpha;
const float* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const float* x;
float beta;
float* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: oneapi::mkl::index_base descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: oneapi::mkl::index_base descr1 , descr2 ;
  //CHECK-NEXT:oneapi::mkl::index_base descr3 ;
  //CHECK-NEXT: dpct::queue_ptr s;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;
  hipStream_t s;

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_HOST;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  constexpr int zero = 0;
  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: int type0 = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatDiagType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatFillMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatDiagType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: diag0 = (oneapi::mkl::diag)0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatFillMode was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: fill0 = (oneapi::mkl::uplo)0;
  //CHECK-NEXT: base0 = descrA;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1027:{{[0-9]+}}: The call to hipsparseGetMatType was replaced with 0 because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: type0 = 0;
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: handle = s;
  //CHECK-NEXT: s = handle;
  hipsparseCreate(&handle);
  hipsparseSetStream(handle,s);
  hipsparseGetStream(handle,&s);

  //CHECK: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetMatType was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: descrA = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, oneapi::mkl::transpose::nontrans, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseScsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipComplex alpha_C, beta_C, *csrValA_C, *x_C, *y_C;

  //CHECK: {
  //CHECK-NEXT: auto csrValA_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(csrValA_C);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_C);
  //CHECK-NEXT: auto y_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_C);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, std::complex<float>(alpha_C.x(), alpha_C.y()), mat_handle_ct{{[0-9]+}}, x_C_buf_ct{{[0-9]+}}, std::complex<float>(beta_C.x(), beta_C.y()), y_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCcsrmv(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, n, ldb, beta, y_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseScsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: {
  //CHECK-NEXT: auto csrValA_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(csrValA_C);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_C);
  //CHECK-NEXT: auto y_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_C);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, k, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, transA, std::complex<float>(alpha_C.x(), alpha_C.y()), mat_handle_ct{{[0-9]+}}, x_C_buf_ct{{[0-9]+}}, n, ldb, std::complex<float>(beta_C.x(), beta_C.y()), y_C_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCcsrmm(handle, transA, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrA, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: int info;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCreateSolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseScsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroySolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseScsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCcsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseCcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(oneapi::mkl::index_base descrB) try {
//CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
//CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
//CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
//CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
//CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
//CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

//CHECK: void foo2(oneapi::mkl::index_base descrB){
//CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
//CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
//CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
//CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
//CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
//CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(mat_handle_ct{{[0-9]+}}, m, n, descrB, csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: }
void foo2(hipsparseMatDescr_t descrB){
  hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

