#include "hip/hip_runtime.h"
// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1, v12.2
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1, cuda-12.2
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cusparse %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cusparse/cusparse.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
float alpha;
const float* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const float* x;
float beta;
float* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> descr1 = 0, descr2 = 0;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::matrix_info> descr3 = 0;
  //CHECK-NEXT: dpct::queue_ptr s;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;
  hipStream_t s;

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_HOST;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  constexpr int zero = 0;
  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: dpct::sparse::matrix_info::matrix_type type0 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: descrA->set_diag((oneapi::mkl::diag)zero);
  //CHECK-NEXT: descrA->set_uplo((oneapi::mkl::uplo)zero);
  //CHECK-NEXT: descrA->set_index_base((oneapi::mkl::index_base)zero);
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)zero);
  //CHECK-NEXT: diag0 = descrA->get_diag();
  //CHECK-NEXT: fill0 = descrA->get_uplo();
  //CHECK-NEXT: base0 = descrA->get_index_base();
  //CHECK-NEXT: type0 = descrA->get_matrix_type();
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_out_of_order_queue();
  //CHECK-NEXT: handle = s;
  //CHECK-NEXT: s = handle;
  hipsparseCreate(&handle);
  hipsparseSetStream(handle,s);
  hipsparseGetStream(handle,&s);

  //CHECK: descrA = std::make_shared<dpct::sparse::matrix_info>();
  //CHECK-NEXT: descrA->set_matrix_type(dpct::sparse::matrix_info::matrix_type::ge);
  //CHECK-NEXT: descrA->set_index_base(oneapi::mkl::index_base::zero);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, (oneapi::mkl::transpose)zero, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseScsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipComplex alpha_C, beta_C, *csrValA_C, *x_C, *y_C;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, transA, m, n, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);
  hipsparseCcsrmv(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
  cusparseScsrmv_mp(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, transA, m, n, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);
  cusparseCcsrmv_mp(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);
  hipsparseScsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);
  hipsparseCcsrmm(handle, transA, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  hipsparseOperation_t transB = HIPSPARSE_OPERATION_NON_TRANSPOSE;
  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, transB, m, n, k, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);
  hipsparseScsrmm2(handle, transA, transB, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, transB, m, n, k, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);
  hipsparseCcsrmm2(handle, transA, transB, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y))){}
  if(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y));;){}
  for(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y))){}
  switch(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: std::shared_ptr<dpct::sparse::optimize_info> info;
  //CHECK-NEXT: info = std::make_shared<dpct::sparse::optimize_info>();
  //CHECK-NEXT: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  //CHECK-NEXT: info.reset();
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseScsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);
  cusparseCcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(std::shared_ptr<dpct::sparse::matrix_info> descrB) try {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
//CHECK-NEXT:  */
//CHECK-NEXT:  return DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y));
//CHECK-NEXT:}
int foo(hipsparseMatDescr_t descrB){
  return hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

//CHECK: void foo2(std::shared_ptr<dpct::sparse::matrix_info> descrB){
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
//CHECK-NEXT:}
void foo2(hipsparseMatDescr_t descrB){
  hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

void foo3(){
  int c_nnz;
  hipsparseMatDescr_t descrB;
  hipsparseMatDescr_t descrC;

  const float* val_a_s;
  const double* val_a_d;
  const float2* val_a_c;
  const double2* val_a_z;
  const int* row_ptr_a;
  const int* col_ind_a;

  const float* val_b_s;
  const double* val_b_d;
  const float2* val_b_c;
  const double2* val_b_z;

  float* val_c_s;
  double* val_c_d;
  float2* val_c_c;
  double2* val_c_z;

  const float* alpha_s;
  const double* alpha_d;
  const float2* alpha_c;
  const double2* alpha_z;

  const float* beta_s;
  const double* beta_d;
  const float2* beta_c;
  const double2* beta_z;

  // CHECK: /*
  // CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::sparse::csrmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 2, 5, alpha_s, descrA, val_a_s, row_ptr_a, col_ind_a, val_b_s, 5, beta_s, val_c_s, 4);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::sparse::csrmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 2, 5, alpha_d, descrA, val_a_d, row_ptr_a, col_ind_a, val_b_d, 5, beta_d, val_c_d, 4);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::sparse::csrmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 2, 5, alpha_c, descrA, val_a_c, row_ptr_a, col_ind_a, val_b_c, 5, beta_c, val_c_c, 4);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::sparse::csrmm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 4, 2, 5, alpha_z, descrA, val_a_z, row_ptr_a, col_ind_a, val_b_z, 5, beta_z, val_c_z, 4);
  hipsparseScsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, alpha_s, descrA, val_a_s, row_ptr_a, col_ind_a, val_b_s, 5, beta_s, val_c_s, 4);
  hipsparseDcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, alpha_d, descrA, val_a_d, row_ptr_a, col_ind_a, val_b_d, 5, beta_d, val_c_d, 4);
  hipsparseCcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, alpha_c, descrA, val_a_c, row_ptr_a, col_ind_a, val_b_c, 5, beta_c, val_c_c, 4);
  hipsparseZcsrmm2(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 4, 2, 5, 9, alpha_z, descrA, val_a_z, row_ptr_a, col_ind_a, val_b_z, 5, beta_z, val_c_z, 4);
}

void foo4(){
  int c_nnz;
  hipsparseMatDescr_t descrB;
  hipsparseMatDescr_t descrC;

  const float* val_a_s;
  const double* val_a_d;
  const float2* val_a_c;
  const double2* val_a_z;
  const int* row_ptr_a;
  const int* col_ind_a;

  const float* val_b_s;
  const double* val_b_d;
  const float2* val_b_c;
  const double2* val_b_z;
  const int* row_ptr_b;
  const int* col_ind_b;

  float* val_c_s;
  double* val_c_d;
  float2* val_c_c;
  double2* val_c_z;
  int* row_ptr_c;
  int* col_ind_c;

  // CHECK: dpct::sparse::csrgemm_nnz_estimate(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 3, 3, 4, descrA, 4, row_ptr_a, col_ind_a, descrB, 5, row_ptr_b, col_ind_b, descrC, row_ptr_c, &c_nnz);
  // CHECK-NEXT: dpct::sparse::csrgemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 3, 3, 4, descrA, val_a_s, row_ptr_a, col_ind_a, descrB, val_b_s, row_ptr_b, col_ind_b, descrC, val_c_s, row_ptr_c, col_ind_c);
  // CHECK-NEXT: dpct::sparse::csrgemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 3, 3, 4, descrA, val_a_d, row_ptr_a, col_ind_a, descrB, val_b_d, row_ptr_b, col_ind_b, descrC, val_c_d, row_ptr_c, col_ind_c);
  // CHECK-NEXT: dpct::sparse::csrgemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 3, 3, 4, descrA, val_a_c, row_ptr_a, col_ind_a, descrB, val_b_c, row_ptr_b, col_ind_b, descrC, val_c_c, row_ptr_c, col_ind_c);
  // CHECK-NEXT: dpct::sparse::csrgemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, 3, 3, 4, descrA, val_a_z, row_ptr_a, col_ind_a, descrB, val_b_z, row_ptr_b, col_ind_b, descrC, val_c_z, row_ptr_c, col_ind_c);
  hipsparseXcsrgemmNnz(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 3, 4, descrA, 4, row_ptr_a,col_ind_a, descrB, 5, row_ptr_b, col_ind_b, descrC, row_ptr_c, &c_nnz);
  hipsparseScsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 3, 4, descrA, 4, val_a_s, row_ptr_a, col_ind_a, descrB, 5, val_b_s, row_ptr_b, col_ind_b, descrC, val_c_s, row_ptr_c, col_ind_c);
  hipsparseDcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 3, 4, descrA, 4, val_a_d, row_ptr_a, col_ind_a, descrB, 5, val_b_d, row_ptr_b, col_ind_b, descrC, val_c_d, row_ptr_c, col_ind_c);
  hipsparseCcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 3, 4, descrA, 4, val_a_c, row_ptr_a, col_ind_a, descrB, 5, val_b_c, row_ptr_b, col_ind_b, descrC, val_c_c, row_ptr_c, col_ind_c);
  hipsparseZcsrgemm(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 3, 4, descrA, 4, val_a_z, row_ptr_a, col_ind_a, descrB, 5, val_b_z, row_ptr_b, col_ind_b, descrC, val_c_z, row_ptr_c, col_ind_c);
}

void foo5() {
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descrA;

  //CHECK:std::shared_ptr<dpct::sparse::optimize_info> info;
  //CHECK-NEXT:info = std::make_shared<dpct::sparse::optimize_info>();
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);

  float *a_s_val;
  double *a_d_val;
  float2 *a_c_val;
  double2 *a_z_val;
  int *a_row_ptr;
  int *a_col_ind;
  float *f_s;
  double *f_d;
  float2 *f_c;
  double2 *f_z;
  float *x_s;
  double *x_d;
  float2 *x_c;
  double2 *x_z;


  float alpha_s = 1;
  double alpha_d = 1;
  float2 alpha_c = float2{1, 0};
  double2 alpha_z = double2{1, 0};

  //CHECK:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_s, descrA, a_s_val, a_row_ptr, a_col_ind, info, f_s, x_s);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_d, descrA, a_d_val, a_row_ptr, a_col_ind, info, f_d, x_d);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_c, descrA, a_c_val, a_row_ptr, a_col_ind, info, f_c, x_c);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_z, descrA, a_z_val, a_row_ptr, a_col_ind, info, f_z, x_z);
  //CHECK-NEXT:info.reset();
  cusparseScsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, &alpha_s, descrA, a_s_val, a_row_ptr, a_col_ind, info, f_s, x_s);
  cusparseDcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, &alpha_d, descrA, a_d_val, a_row_ptr, a_col_ind, info, f_d, x_d);
  cusparseCcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, &alpha_c, descrA, a_c_val, a_row_ptr, a_col_ind, info, f_c, x_c);
  cusparseZcsrsv_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, &alpha_z, descrA, a_z_val, a_row_ptr, a_col_ind, info, f_z, x_z);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK:std::shared_ptr<dpct::sparse::optimize_info> info2;
  //CHECK-NEXT:info2 = std::make_shared<dpct::sparse::optimize_info>();
  //CHECK-NEXT:int policy = 1;
  //CHECK-NEXT:policy = 0;
  csrsv2Info_t info2;
  hipsparseCreateCsrsv2Info(&info2);
  hipsparseSolvePolicy_t policy = HIPSPARSE_SOLVE_POLICY_USE_LEVEL;
  policy = HIPSPARSE_SOLVE_POLICY_NO_LEVEL;

  int buffer_size_s0;
  int buffer_size_d0;
  int buffer_size_c0;
  int buffer_size_z0;
  //CHECK:buffer_size_s0 = 0;
  //CHECK-NEXT:buffer_size_d0 = 0;
  //CHECK-NEXT:buffer_size_c0 = 0;
  //CHECK-NEXT:buffer_size_z0 = 0;
  hipsparseScsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_s_val, a_row_ptr, a_col_ind, info2, &buffer_size_s0);
  hipsparseDcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_d_val, a_row_ptr, a_col_ind, info2, &buffer_size_d0);
  hipsparseCcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_c_val, a_row_ptr, a_col_ind, info2, &buffer_size_c0);
  hipsparseZcsrsv2_bufferSize(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_z_val, a_row_ptr, a_col_ind, info2, &buffer_size_z0);

  size_t buffer_size_s;
  size_t buffer_size_d;
  size_t buffer_size_c;
  size_t buffer_size_z;
  //CHECK:buffer_size_s = 0;
  //CHECK-NEXT:buffer_size_d = 0;
  //CHECK-NEXT:buffer_size_c = 0;
  //CHECK-NEXT:buffer_size_z = 0;
  hipsparseScsrsv2_bufferSizeExt(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_s_val, a_row_ptr, a_col_ind, info2, &buffer_size_s);
  hipsparseDcsrsv2_bufferSizeExt(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_d_val, a_row_ptr, a_col_ind, info2, &buffer_size_d);
  hipsparseCcsrsv2_bufferSizeExt(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_c_val, a_row_ptr, a_col_ind, info2, &buffer_size_c);
  hipsparseZcsrsv2_bufferSizeExt(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_z_val, a_row_ptr, a_col_ind, info2, &buffer_size_z);

  void* buffer_s;
  void* buffer_d;
  void* buffer_c;
  void* buffer_z;
  hipMalloc(&buffer_s, buffer_size_s);
  hipMalloc(&buffer_d, buffer_size_d);
  hipMalloc(&buffer_c, buffer_size_c);
  hipMalloc(&buffer_z, buffer_size_z);

  //CHECK:dpct::sparse::optimize_csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, descrA, a_s_val, a_row_ptr, a_col_ind, info2);
  //CHECK-NEXT:dpct::sparse::optimize_csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, descrA, a_d_val, a_row_ptr, a_col_ind, info2);
  //CHECK-NEXT:dpct::sparse::optimize_csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, descrA, a_c_val, a_row_ptr, a_col_ind, info2);
  //CHECK-NEXT:dpct::sparse::optimize_csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, descrA, a_z_val, a_row_ptr, a_col_ind, info2);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_s, descrA, a_s_val, a_row_ptr, a_col_ind, info2, f_s, x_s);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_d, descrA, a_d_val, a_row_ptr, a_col_ind, info2, f_d, x_d);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_c, descrA, a_c_val, a_row_ptr, a_col_ind, info2, f_c, x_c);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_z, descrA, a_z_val, a_row_ptr, a_col_ind, info2, f_z, x_z);
  //CHECK-NEXT:info2.reset();
  hipsparseScsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_s_val, a_row_ptr, a_col_ind, info2, policy, buffer_s);
  hipsparseDcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_d_val, a_row_ptr, a_col_ind, info2, policy, buffer_d);
  hipsparseCcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_c_val, a_row_ptr, a_col_ind, info2, policy, buffer_c);
  hipsparseZcsrsv2_analysis(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_z_val, a_row_ptr, a_col_ind, info2, policy, buffer_z);
  hipsparseScsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, &alpha_s, descrA, a_s_val, a_row_ptr, a_col_ind, info2, f_s, x_s, policy, buffer_s);
  hipsparseDcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, &alpha_d, descrA, a_d_val, a_row_ptr, a_col_ind, info2, f_d, x_d, policy, buffer_d);
  hipsparseCcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, &alpha_c, descrA, a_c_val, a_row_ptr, a_col_ind, info2, f_c, x_c, policy, buffer_c);
  hipsparseZcsrsv2_solve(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, &alpha_z, descrA, a_z_val, a_row_ptr, a_col_ind, info2, f_z, x_z, policy, buffer_z);
  hipsparseDestroyCsrsv2Info(info2);
}

void foo6() {
  hipsparseHandle_t handle;
  hipsparseMatDescr_t descrA;
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);

  float *a_s_val;
  int *a_row_ptr;
  int *a_col_ind;
  float *f_s;
  float *x_s;
  float alpha_s = 1;

  //CHECK:dpct::sparse::optimize_csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, descrA, a_s_val, dpct::library_data_t::real_float, a_row_ptr, a_col_ind, info);
  //CHECK-NEXT:dpct::sparse::csrsv(*handle, oneapi::mkl::transpose::nontrans, 3, &alpha_s, dpct::library_data_t::real_float, descrA, a_s_val, dpct::library_data_t::real_float, a_row_ptr, a_col_ind, info, f_s, dpct::library_data_t::real_float, x_s, dpct::library_data_t::real_float);
  cusparseCsrsv_analysisEx(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, 6, descrA, a_s_val, HIP_R_32F, a_row_ptr, a_col_ind, info, HIP_R_32F);
  cusparseCsrsv_solveEx(handle, HIPSPARSE_OPERATION_NON_TRANSPOSE, 3, &alpha_s, HIP_R_32F, descrA, a_s_val, HIP_R_32F, a_row_ptr, a_col_ind, info, f_s, HIP_R_32F, x_s, HIP_R_32F, HIP_R_32F);
}
