// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cusparse %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cusparse/cusparse.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
float alpha;
const float* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const float* x;
float beta;
float* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> descr1 = 0, descr2 = 0;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::matrix_info> descr3 = 0;
  //CHECK-NEXT: dpct::queue_ptr s;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;
  hipStream_t s;

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_HOST;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  constexpr int zero = 0;
  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: dpct::sparse::matrix_info::matrix_type type0 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: descrA->set_diag((oneapi::mkl::diag)zero);
  //CHECK-NEXT: descrA->set_uplo((oneapi::mkl::uplo)zero);
  //CHECK-NEXT: descrA->set_index_base((oneapi::mkl::index_base)zero);
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)zero);
  //CHECK-NEXT: diag0 = descrA->get_diag();
  //CHECK-NEXT: fill0 = descrA->get_uplo();
  //CHECK-NEXT: base0 = descrA->get_index_base();
  //CHECK-NEXT: type0 = descrA->get_matrix_type();
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_default_queue();
  //CHECK-NEXT: handle = s;
  //CHECK-NEXT: s = handle;
  hipsparseCreate(&handle);
  hipsparseSetStream(handle,s);
  hipsparseGetStream(handle,&s);

  //CHECK: descrA = std::make_shared<dpct::sparse::matrix_info>();
  //CHECK-NEXT: descrA->set_matrix_type(dpct::sparse::matrix_info::matrix_type::ge);
  //CHECK-NEXT: descrA->set_index_base(oneapi::mkl::index_base::zero);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, oneapi::mkl::transpose::nontrans, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseScsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipComplex alpha_C, beta_C, *csrValA_C, *x_C, *y_C;

  //CHECK: {
  //CHECK-NEXT: auto csrValA_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(csrValA_C);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_C);
  //CHECK-NEXT: auto y_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_C);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, std::complex<float>(alpha_C.x(), alpha_C.y()), mat_handle_ct{{[0-9]+}}, x_C_buf_ct{{[0-9]+}}, std::complex<float>(beta_C.x(), beta_C.y()), y_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCcsrmv(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, k, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, oneapi::mkl::layout::row_major, transA, oneapi::mkl::transpose::nontrans, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, n, ldb, beta, y_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseScsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: {
  //CHECK-NEXT: auto csrValA_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(csrValA_C);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(x_C);
  //CHECK-NEXT: auto y_C_buf_ct{{[0-9]+}} = dpct::get_buffer<std::complex<float>>(y_C);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, k, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_C_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemm(*handle, oneapi::mkl::layout::row_major, transA, oneapi::mkl::transpose::nontrans, std::complex<float>(alpha_C.x(), alpha_C.y()), mat_handle_ct{{[0-9]+}}, x_C_buf_ct{{[0-9]+}}, n, ldb, std::complex<float>(beta_C.x(), beta_C.y()), y_C_buf_ct{{[0-9]+}}, ldc);
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  hipsparseCcsrmm(handle, transA, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = 0){}
  if(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a for statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = 0;;){}
  for(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: {
  //CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
  //CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
  //CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
  //CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
  //CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
  //CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
  //CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrA->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
  //CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
  //CHECK-NEXT: }
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a switch statement. You may need to rewrite this code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = 0){}
  switch(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: int info;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCreateSolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseScsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseDestroySolveAnalysisInfo was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseScsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cusparseCcsrsv_analysis was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  cusparseCcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(std::shared_ptr<dpct::sparse::matrix_info> descrB) try {
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
//CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
//CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
//CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
//CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
//CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrB->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
//CHECK-NEXT: */
//CHECK-NEXT: return 0;
//CHECK-NEXT: }
int foo(hipsparseMatDescr_t descrB){
  return hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

//CHECK: void foo2(std::shared_ptr<dpct::sparse::matrix_info> descrB){
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: auto csrValA_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(csrValA);
//CHECK-NEXT: auto csrRowPtrA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrRowPtrA);
//CHECK-NEXT: auto csrColIndA_buf_ct{{[0-9]+}} = dpct::get_buffer<int>(csrColIndA);
//CHECK-NEXT: auto x_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(x);
//CHECK-NEXT: auto y_buf_ct{{[0-9]+}} = dpct::get_buffer<float>(y);
//CHECK-NEXT: oneapi::mkl::sparse::matrix_handle_t mat_handle_ct{{[0-9]+}};
//CHECK-NEXT: oneapi::mkl::sparse::init_matrix_handle(&mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::set_csr_data(*handle, mat_handle_ct{{[0-9]+}}, m, n, descrB->get_index_base(), csrRowPtrA_buf_ct{{[0-9]+}}, csrColIndA_buf_ct{{[0-9]+}}, csrValA_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::gemv(*handle, transA, alpha, mat_handle_ct{{[0-9]+}}, x_buf_ct{{[0-9]+}}, beta, y_buf_ct{{[0-9]+}});
//CHECK-NEXT: oneapi::mkl::sparse::release_matrix_handle(*handle, &mat_handle_ct{{[0-9]+}});
//CHECK-NEXT: }
void foo2(hipsparseMatDescr_t descrB){
  hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

