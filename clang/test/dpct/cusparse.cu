// UNSUPPORTED: v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1, v12.2
// UNSUPPORTED: cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1, cuda-12.2
// RUN: dpct --format-range=none --usm-level=none --out-root %T/cusparse %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only --std=c++14
// RUN: FileCheck --input-file %T/cusparse/cusparse.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipsparse.h>
#include <hip/hip_runtime.h>

int m, n, nnz, k, ldb, ldc;
float alpha;
const float* csrValA;
const int* csrRowPtrA;
const int* csrColIndA;
const float* x;
float beta;
float* y;
//CHECK: sycl::queue* handle;
//CHECK-NEXT: oneapi::mkl::transpose transA = oneapi::mkl::transpose::nontrans;
//CHECK-NEXT: std::shared_ptr<dpct::sparse::matrix_info> descrA;
hipsparseHandle_t handle;
hipsparseOperation_t transA = HIPSPARSE_OPERATION_NON_TRANSPOSE;
hipsparseMatDescr_t descrA;

int main(){
  //CHECK: std::shared_ptr<dpct::sparse::matrix_info> descr1 = 0, descr2 = 0;
  //CHECK-NEXT:std::shared_ptr<dpct::sparse::matrix_info> descr3 = 0;
  //CHECK-NEXT: dpct::queue_ptr s;
  hipsparseMatDescr_t descr1 = 0, descr2 = 0;
  hipsparseMatDescr_t descr3 = 0;
  hipStream_t s;

  //CHECK: int mode = 0;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseGetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseSetPointerMode was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  hipsparsePointerMode_t mode = HIPSPARSE_POINTER_MODE_HOST;
  hipsparseGetPointerMode(handle, &mode);
  hipsparseSetPointerMode(handle, HIPSPARSE_POINTER_MODE_DEVICE);

  constexpr int zero = 0;
  //CHECK: oneapi::mkl::diag diag0 = oneapi::mkl::diag::nonunit;
  //CHECK-NEXT: oneapi::mkl::uplo fill0 = oneapi::mkl::uplo::lower;
  //CHECK-NEXT: oneapi::mkl::index_base base0 = oneapi::mkl::index_base::zero;
  //CHECK-NEXT: dpct::sparse::matrix_info::matrix_type type0 = dpct::sparse::matrix_info::matrix_type::ge;
  //CHECK-NEXT: descrA->set_diag((oneapi::mkl::diag)zero);
  //CHECK-NEXT: descrA->set_uplo((oneapi::mkl::uplo)zero);
  //CHECK-NEXT: descrA->set_index_base((oneapi::mkl::index_base)zero);
  //CHECK-NEXT: descrA->set_matrix_type((dpct::sparse::matrix_info::matrix_type)zero);
  //CHECK-NEXT: diag0 = descrA->get_diag();
  //CHECK-NEXT: fill0 = descrA->get_uplo();
  //CHECK-NEXT: base0 = descrA->get_index_base();
  //CHECK-NEXT: type0 = descrA->get_matrix_type();
  hipsparseDiagType_t diag0 = HIPSPARSE_DIAG_TYPE_NON_UNIT;
  hipsparseFillMode_t fill0 = HIPSPARSE_FILL_MODE_LOWER;
  hipsparseIndexBase_t base0 = HIPSPARSE_INDEX_BASE_ZERO;
  hipsparseMatrixType_t type0 = HIPSPARSE_MATRIX_TYPE_GENERAL;
  hipsparseSetMatDiagType(descrA, (hipsparseDiagType_t)zero);
  hipsparseSetMatFillMode(descrA, (hipsparseFillMode_t)zero);
  hipsparseSetMatIndexBase(descrA, (hipsparseIndexBase_t)zero);
  hipsparseSetMatType(descrA, (hipsparseMatrixType_t)zero);
  diag0 = hipsparseGetMatDiagType(descrA);
  fill0 = hipsparseGetMatFillMode(descrA);
  base0 = hipsparseGetMatIndexBase(descrA);
  type0 = hipsparseGetMatType(descrA);

  //CHECK: handle = &dpct::get_out_of_order_queue();
  //CHECK-NEXT: handle = s;
  //CHECK-NEXT: s = handle;
  hipsparseCreate(&handle);
  hipsparseSetStream(handle,s);
  hipsparseGetStream(handle,&s);

  //CHECK: descrA = std::make_shared<dpct::sparse::matrix_info>();
  //CHECK-NEXT: descrA->set_matrix_type(dpct::sparse::matrix_info::matrix_type::ge);
  //CHECK-NEXT: descrA->set_index_base(oneapi::mkl::index_base::zero);
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, (oneapi::mkl::transpose)zero, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
  hipsparseCreateMatDescr(&descrA);
  hipsparseSetMatType(descrA, HIPSPARSE_MATRIX_TYPE_GENERAL);
  hipsparseSetMatIndexBase(descrA, HIPSPARSE_INDEX_BASE_ZERO);
  hipsparseScsrmv(handle, (hipsparseOperation_t)zero, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);

  hipComplex alpha_C, beta_C, *csrValA_C, *x_C, *y_C;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmv(*handle, transA, m, n, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);
  hipsparseCcsrmv(handle, transA, m, n, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, &beta_C, y_C);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);
  hipsparseScsrmm(handle, transA, m, n, k, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, ldb, &beta, y, ldc);

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: dpct::sparse::csrmm(*handle, transA, m, n, k, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);
  hipsparseCcsrmm(handle, transA, m, n, k, nnz, &alpha_C, descrA, csrValA_C, csrRowPtrA, csrColIndA, x_C, ldb, &beta_C, y_C, ldc);

  //CHECK:int status;
  hipsparseStatus_t status;

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: if(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y))){}
  if(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: for(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y));;){}
  for(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);;){}

  //CHECK: /*
  //CHECK-NEXT: DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: switch(status = DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y))){}
  switch(status = hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrA, csrValA, csrRowPtrA, csrColIndA, x, &beta, y)){}

  //CHECK: std::shared_ptr<dpct::sparse::optimize_info> info;
  //CHECK-NEXT: info = std::make_shared<dpct::sparse::optimize_info>();
  //CHECK-NEXT: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  //CHECK-NEXT: info.reset();
  cusparseSolveAnalysisInfo_t info;
  cusparseCreateSolveAnalysisInfo(&info);
  cusparseScsrsv_analysis(handle, transA, m, nnz, descrA, csrValA, csrRowPtrA, csrColIndA, info);
  cusparseDestroySolveAnalysisInfo(info);

  //CHECK: dpct::sparse::optimize_csrsv(*handle, transA, m, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);
  cusparseCcsrsv_analysis(handle, transA, m, nnz, descrA, csrValA_C, csrRowPtrA, csrColIndA, info);

  //CHECK: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to hipsparseDestroyMatDescr was removed because this call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: handle = nullptr;
  hipsparseDestroyMatDescr(descrA);
  hipsparseDestroy(handle);
}

//CHECK: int foo(std::shared_ptr<dpct::sparse::matrix_info> descrB) try {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
//CHECK-NEXT:  */
//CHECK-NEXT:  return DPCT_CHECK_ERROR(dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y));
//CHECK-NEXT:}
int foo(hipsparseMatDescr_t descrB){
  return hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

//CHECK: void foo2(std::shared_ptr<dpct::sparse::matrix_info> descrB){
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1045:{{[0-9]+}}: Migration is only supported for this API for the general/symmetric/triangular sparse matrix type. You may need to adjust the code.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::sparse::csrmv(*handle, transA, m, n, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
//CHECK-NEXT:}
void foo2(hipsparseMatDescr_t descrB){
  hipsparseScsrmv(handle, transA, m, n, nnz, &alpha, descrB, csrValA, csrRowPtrA, csrColIndA, x, &beta, y);
}

