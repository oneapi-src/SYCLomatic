// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none -out-root %T/warp %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/warp/warp.dp.cpp --match-full-lines %s

#include "hip/hip_runtime.h"


#define FULL_MASK 23
#define NUM_ELEMENTS 1024

__global__ void kernel1() {
  int predicate;
  // CHECK: sycl::all_of_group(item_{{[0-9a-z]+}}.get_sub_group(), predicate);
  __all(predicate);
}

__global__ void kernel2() {
  int predicate;
  //CHECK: /*
  //CHECK-NEXT: DPCT1086:{{[0-9]+}}: __activemask() is migrated to 0xffffffff. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::all_of_group(item_{{[0-9a-z]+}}.get_sub_group(), (~0xffffffff & (0x1 << item_{{[0-9a-z]+}}.get_sub_group().get_local_linear_id())) || predicate);
  __all_sync(__activemask(), predicate);
}

__global__ void kernel3() {
  int predicate;
  unsigned mask;
  // CHECK: sycl::all_of_group(item_{{[0-9a-z]+}}.get_sub_group(), (~mask & (0x1 << item_{{[0-9a-z]+}}.get_sub_group().get_local_linear_id())) || predicate);
  __all_sync(mask, predicate);
}

__global__ void kernel4() {
  int predicate;
  // CHECK: sycl::any_of_group(item_{{[0-9a-z]+}}.get_sub_group(), predicate);
  __any(predicate);
}

__global__ void kernel5() {
  int predicate;
  //CHECK: /*
  //CHECK-NEXT: DPCT1086:{{[0-9]+}}: __activemask() is migrated to 0xffffffff. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: sycl::any_of_group(item_ct1.get_sub_group(), (0xffffffff & (0x1 << item_ct1.get_sub_group().get_local_linear_id())) && predicate);
  __any_sync(__activemask(), predicate);
}

__global__ void kernel6() {
  int predicate;
  unsigned mask;
  // CHECK: sycl::any_of_group(item_{{[0-9a-z]+}}.get_sub_group(), (mask & (0x1 << item_{{[0-9a-z]+}}.get_sub_group().get_local_linear_id())) && predicate);
  __any_sync(mask, predicate);
}

__global__ void kernel7() {
  int val;
  int srcLane;
  // CHECK: /*
  // CHECK: DPCT1096:{{[0-9]+}}: The right-most dimension of the work-group used in the SYCL kernel that calls this function may be less than "32". The function "dpct::select_from_sub_group" may return an unexpected result on the CPU device. Modify the size of the work-group to ensure that the value of the right-most dimension is a multiple of "32".
  // CHECK: */
  // CHECK: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane);
  __shfl(val, srcLane);
}

__global__ void kernel8() {
  unsigned mask;
  int val;
  int srcLane;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::select_from_sub_group. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1096:{{[0-9]+}}: The right-most dimension of the work-group used in the SYCL kernel that calls this function may be less than "32". The function "dpct::select_from_sub_group" may return an unexpected result on the CPU device. Modify the size of the work-group to ensure that the value of the right-most dimension is a multiple of "32".
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane);
  __shfl_sync(mask, val, srcLane);
}

__global__ void kernel9() {
  unsigned mask;
  int val;
  int srcLane;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::select_from_sub_group. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1096:{{[0-9]+}}: The right-most dimension of the work-group used in the SYCL kernel that calls this function may be less than "32". The function "dpct::select_from_sub_group" may return an unexpected result on the CPU device. Modify the size of the work-group to ensure that the value of the right-most dimension is a multiple of "32".
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane);
  __shfl_sync(mask, val, srcLane, warpSize);
}

__global__ void kernel10() {
  unsigned delta;
  int val;
  // CHECK: dpct::shift_sub_group_right(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_up(val, delta);
}

__global__ void kernel11() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT:DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_right. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_up_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_right(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_up_sync(mask, val, delta);
}

__global__ void kernel12() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT:DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_right. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_up_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_right(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_up_sync(mask, val, delta, warpSize);
}

__global__ void kernel13() {
  int val;
  unsigned delta;
  // CHECK: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_down(val, delta);
}

__global__ void kernel14() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_left. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_down_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_down_sync(mask, val, delta);
}

__global__ void kernel15() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_left. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_down_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, delta);
  __shfl_down_sync(mask, val, delta, warpSize);
}

__global__ void kernel16() {
  int laneMask;
  int val;
  // CHECK: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask);
  __shfl_xor(val, laneMask);
}

__global__ void kernel17() {
  unsigned mask;
  int val;
  int laneMask;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask);
  __shfl_xor_sync(mask, val, laneMask);
}

__global__ void kernel18() {
  unsigned mask;
  int val;
  int laneMask;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask);
  __shfl_xor_sync(mask, val, laneMask, warpSize);
}

__global__ void kernel19() {
  unsigned mask;
  // CHECK: mask = sycl::reduce_over_group(item_ct{{[0-9a-z]+}}.get_sub_group(), item_ct{{[0-9a-z]+}}.get_local_id(2) < NUM_ELEMENTS ? (0x1 << item_ct{{[0-9a-z]+}}.get_sub_group().get_local_linear_id()) : 0, sycl::ext::oneapi::plus<>());
  mask = __ballot(threadIdx.x < NUM_ELEMENTS);
}

__global__ void kernel20() {
  unsigned mask;
  // CHECK: mask = sycl::reduce_over_group(item_ct{{[0-9a-z]+}}.get_sub_group(), (0xffffffff & (0x1 << item_ct{{[0-9a-z]+}}.get_sub_group().get_local_linear_id())) && item_ct{{[0-9a-z]+}}.get_local_id(2) < NUM_ELEMENTS ? (0x1 << item_ct{{[0-9a-z]+}}.get_sub_group().get_local_linear_id()) : 0, sycl::ext::oneapi::plus<>());
  mask = __ballot_sync(__activemask(), threadIdx.x < NUM_ELEMENTS);
}

__global__ void kernel21() {
  unsigned mask;
  // CHECK: mask = sycl::reduce_over_group(item_ct{{[0-9a-z]+}}.get_sub_group(), (FULL_MASK & (0x1 << item_ct{{[0-9a-z]+}}.get_sub_group().get_local_linear_id())) && item_ct{{[0-9a-z]+}}.get_local_id(2) < NUM_ELEMENTS ? (0x1 << item_ct{{[0-9a-z]+}}.get_sub_group().get_local_linear_id()) : 0, sycl::ext::oneapi::plus<>());
  mask = __ballot_sync(FULL_MASK, threadIdx.x < NUM_ELEMENTS);
}

__global__ void kernel22() {
  unsigned mask;
  //CHECK: /*
  //CHECK-NEXT: DPCT1086:{{[0-9]+}}: __activemask() is migrated to 0xffffffff. You may need to adjust the code.
  //CHECK-NEXT: */
  //CHECK-NEXT: mask = 0xffffffff;
  mask = __activemask();
}

__global__ void kernel23() {
  int val;
  int srcLane;
  // CHECK: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane, 16);
  __shfl(val, srcLane, 16);
}

__global__ void kernel24() {
  unsigned mask;
  int val;
  int srcLane;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::select_from_sub_group. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane, 16);
  __shfl_sync(mask, val, srcLane, 16);
}

__global__ void kernel25() {
  int val;
  unsigned delta;
  // CHECK: dpct::shift_sub_group_right(item_{{[0-9a-z]+}}.get_sub_group(), val, delta, 16);
  __shfl_up(val, delta, 16);
}

__global__ void kernel26() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT:DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_right. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_up_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_right(item_{{[0-9a-z]+}}.get_sub_group(), val, delta, 16);
  __shfl_up_sync(mask, val, delta, 16);
}

__global__ void kernel27() {
  int val;
  unsigned delta;
  // CHECK: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, delta, 16);
  __shfl_down(val, delta, 16);
}

__global__ void kernel28() {
  unsigned mask;
  int val;
  unsigned delta;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_left. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_down_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, delta, 16);
  __shfl_down_sync(mask, val, delta, 16);
}

__global__ void kernel29() {
  int val;
  int laneMask;
  // CHECK: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, 16);
  __shfl_xor(val, laneMask, 16);
}

__global__ void kernel30() {
  unsigned mask;
  int val;
  int laneMask;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, 16);
  __shfl_xor_sync(mask, val, laneMask, 16);
}

__global__ void kernel31() {
  unsigned mask;
  int val;
  int srcLane;
  // CHECK: dpct::select_from_sub_group(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane, 16);
  __shfl(val, srcLane, 16);
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::shift_sub_group_left. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_down_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::shift_sub_group_left(item_{{[0-9a-z]+}}.get_sub_group(), val, srcLane);
  __shfl_down_sync(mask, val, srcLane, 32);
}

__global__ void kernel32() {
  unsigned mask;
  int val;
  int laneMask;
  int warpSize;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, warpSize);
  __shfl_xor_sync(mask, val, laneMask, warpSize);
}

__global__ void kernel33() {
  unsigned mask;
  int val;
  int laneMask;
  int WS;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, WS);
  __shfl_xor_sync(mask, val, laneMask, WS);
}

__global__ void kernel34() {
  int val;
  int laneMask;
  int WS;
  // CHECK: /*
  // CHECK-NEXT: DPCT1023:{{[0-9]+}}: The SYCL sub-group does not support mask options for dpct::permute_sub_group_by_xor. You can specify "--use-experimental-features=masked-sub-group-operation" to use the experimental helper function to migrate __shfl_xor_sync.
  // CHECK-NEXT: */
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, WS);
  __shfl_xor_sync(0x7FFFFFFF, val, laneMask, WS);
}

__global__ void kernel35() {
  // CHECK: int val;
  // CHECK-NEXT: int laneMask;
  // CHECK-NEXT: int WS;
  // CHECK-NEXT: dpct::permute_sub_group_by_xor(item_{{[0-9a-z]+}}.get_sub_group(), val, laneMask, WS);
  int val;
  int laneMask;
  int WS;
  __shfl_xor_sync(0xFFFFFFFF, val, laneMask, WS);
}

int main() {

  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK: sycl::queue &q_ct1 = dev_ct1.in_order_queue();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel1(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel1<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel2(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel2<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel3(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel3<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel4(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel4<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel5(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel5<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel6(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel6<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel7(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel7<<<1,1>>>();

  auto BS = dim3(1);
  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(BS, BS),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel8(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel8<<<1,BS>>>();
  
  int a;
  auto BS2 = dim3(a);
  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(BS2, BS2),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel9(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel9<<<1,BS2>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel10(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel10<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel11(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel11<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel12(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel12<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel13(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel13<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel14(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel14<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel15(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel15<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel16(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel16<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel17(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel17<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel18(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel18<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel19(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel19<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel20(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel20<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel21(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel21<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {
  // CHECK-NEXT:     kernel22();
  // CHECK-NEXT:   });
  kernel22<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel23(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel23<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel24(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel24<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel25(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel25<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel26(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel26<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel27(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel27<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel28(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel28<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel29(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel29<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel30(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel30<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel31(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel31<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel32(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel32<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel33(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel33<<<1,32>>>();

    // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel34(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel34<<<1,32>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 32), sycl::range<3>(1, 1, 32)),
  // CHECK-NEXT:   [=](sycl::nd_item<3> item_{{[0-9a-z]+}}) {{\[\[}}intel::reqd_sub_group_size(32){{\]\]}} {
  // CHECK-NEXT:     kernel35(item_{{[0-9a-z]+}});
  // CHECK-NEXT:   });
  kernel35<<<1,32>>>();
}