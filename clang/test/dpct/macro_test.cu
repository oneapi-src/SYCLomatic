#include "hip/hip_runtime.h"
// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: cat %s > %T/macro_test.cu
// RUN: cat %S/macro_test.h > %T/macro_test.h
// RUN: cd %T
// RUN: rm -rf %T/macro_test_output
// RUN: mkdir %T/macro_test_output
// RUN: dpct -out-root %T/macro_test_output macro_test.cu --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/macro_test_output/macro_test.dp.cpp --match-full-lines macro_test.cu
// RUN: %if build_lit %{icpx -c -fsycl -DBUILD_TEST  %T/macro_test_output/macro_test.dp.cpp -o %T/macro_test_output/macro_test.dp.o %}
// RUN: FileCheck --input-file %T/macro_test_output/macro_test.h --match-full-lines macro_test.h
#ifndef BUILD_TEST
#include "hip/hip_runtime.h"
#include <math.h>
#include <iostream>
#include <cmath>
#include <iomanip>
#include <limits>
#include <algorithm>
#include <hipsolver.h>
#include <stdexcept>

#include <stdio.h>

// CHECK: #include <algorithm>

#include "macro_test.h"

#include <hipblas.h>
#include <thrust/inner_product.h>
#include <thrust/extrema.h>
#include <thrust/host_vector.h>
#include <thrust/gather.h>
#include <thrust/scatter.h>
#include <thrust/tuple.h>
#include <thrust/device_ptr.h>
#include <thrust/copy.h>
#include <thrust/device_vector.h>
#include <thrust/execution_policy.h>
#include <thrust/random.h>
#include <thrust/reduce.h>

#define CUDA_NUM_THREADS 1024+32
#define GET_BLOCKS(n,t)  1+n+t-1
#define GET_BLOCKS2(n,t) 1+n+t
#define GET_BLOCKS3(n,t) n+t-1
#define GET_BLOCKS4(n,t) n+t

#define NESTMACRO(k) k
#define NESTMACRO2(k) NESTMACRO(k)
#define NESTMACRO3(k) NESTMACRO2(k)

class DDD{
public:
  dim3* A;
  dim3 B;
};
#define CALL(x) x;

#define EMPTY_MACRO(x) x
//CHECK:#define GET_MEMBER_MACRO(x) x[1] = 5
#define GET_MEMBER_MACRO(x) x.y = 5

__global__ void foo_kernel() {}

//CHECK: void foo_kernel2(int a, int b
//CHECK-NEXT:   #ifdef MACRO_CC
//CHECK-NEXT:   , int c
//CHECK-NEXT:   #endif
//CHECK-NEXT:   , const sycl::nd_item<3> &item_ct1) {
//CHECK-NEXT:     int x = item_ct1.get_group(2);
//CHECK-NEXT:   }
__global__ void foo_kernel2(int a, int b
#ifdef MACRO_CC
, int c
#endif
) {
  int x = blockIdx.x;
}

__global__ void foo2(){
  // CHECK: #define IMUL(a, b) sycl::mul24(a, b)
  // CHECK-NEXT: int vectorBase = IMUL(1, 2);
  #define IMUL(a, b) __mul24(a, b)
  int vectorBase = IMUL(1, 2);
}

__global__ void foo3(int x, int y) {}

void foo() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK-NEXT: sycl::queue &q_ct1 = dev_ct1.in_order_queue();
  DDD d3;

// CHECK: #ifdef DPCT_COMPATIBILITY_TEMP
#ifdef __CUDA_ARCH__
  // CHECK: int CA = DPCT_COMPATIBILITY_TEMP;
  int CA = __CUDA_ARCH__;
#endif


  // CHECK: (*d3.A)[2] = 3;
  // CHECK-NEXT: d3.B[2] = 2;
  // CHECK-NEXT: EMPTY_MACRO(d3.B[2]);
  // CHECK-NEXT: GET_MEMBER_MACRO(d3.B);
  d3.A->x = 3;
  d3.B.x = 2;
  EMPTY_MACRO(d3.B.x);
  GET_MEMBER_MACRO(d3.B);

  int outputThreadCount = 512;

  //CHECK: /*
  //CHECK-NEXT: DPCT1038:{{[0-9]+}}: When the kernel function name is used as a macro argument, the
  //CHECK-NEXT: migration result may be incorrect. You need to verify the definition of the
  //CHECK-NEXT: macro.
  //CHECK-NEXT: */
  //CHECK-NEXT: CALL(([&]() {
  //CHECK-NEXT:   q_ct1.parallel_for(
  //CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 2), sycl::range<3>(1, 1, 2)),
  //CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:           foo_kernel();
  //CHECK-NEXT:         });
  //CHECK-NEXT: }()))
  CALL( (foo_kernel<<<1, 2, 0>>>()) )

  //CHECK: #define AA 3
  //CHECK-NEXT: #define MCALL                                                                  \
  //CHECK-NEXT: q_ct1.parallel_for(sycl::nd_range<3>(sycl::range<3>(1, 1, 2) *               \
  //CHECK-NEXT:                                          sycl::range<3>(1, 1, 2 * AA),       \
  //CHECK-NEXT:                                      sycl::range<3>(1, 1, 2 * AA)),          \
  //CHECK-NEXT:                    [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
  //CHECK-NEXT: MCALL
  #define AA 3
  #define MCALL foo_kernel<<<dim3(2,1), 2*AA, 0>>>();
  MCALL

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS(outputThreadCount, outputThreadCount)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 2),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 2)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         foo_kernel();
  // CHECK-NEXT:       });
  foo_kernel<<<GET_BLOCKS(outputThreadCount, outputThreadCount), 2, 0>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS2(CUDA_NUM_THREADS, CUDA_NUM_THREADS)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 0),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 0)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         foo_kernel();
  // CHECK-NEXT:       });
  foo_kernel<<<GET_BLOCKS2(CUDA_NUM_THREADS, CUDA_NUM_THREADS), 0, 0>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS3(CUDA_NUM_THREADS, outputThreadCount)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 0),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 0)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         foo_kernel();
  // CHECK-NEXT:       });
  foo_kernel<<<GET_BLOCKS3(CUDA_NUM_THREADS, outputThreadCount), 0, 0>>>();

  // CHECK: q_ct1.parallel_for(
  // CHECK-NEXT:       sycl::nd_range<3>(
  // CHECK-NEXT:           sycl::range<3>(1, 1,
  // CHECK-NEXT:                          GET_BLOCKS4(outputThreadCount, CUDA_NUM_THREADS)) *
  // CHECK-NEXT:               sycl::range<3>(1, 1, 2),
  // CHECK-NEXT:           sycl::range<3>(1, 1, 2)),
  // CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  // CHECK-NEXT:         foo_kernel();
  // CHECK-NEXT:       });
  foo_kernel<<<GET_BLOCKS4(outputThreadCount, CUDA_NUM_THREADS), 2, 0>>>();

  // Test if SIGABRT.
  // No check here because the generated code needs further fine tune.
  #define MACRO_CALL(a, b) foo_kernel<<<a, b, 0>>>();
  MACRO_CALL(0,0)

// CHECK: #define HANDLE_GPU_ERROR(err) \
// CHECK-NEXT: do \
// CHECK-NEXT: { \
// CHECK-NEXT:     if (err != 0) \
// CHECK-NEXT:     { \
// CHECK-NEXT:         int currentDevice; \
// CHECK-NEXT:         currentDevice = dpct::dev_mgr::instance().current_device_id(); \
// CHECK-NEXT:     } \
// CHECK-NEXT: } while (0)
#define HANDLE_GPU_ERROR(err) \
do \
{ \
    if(err != hipSuccess) \
    { \
        int currentDevice; \
        hipGetDevice(&currentDevice); \
    } \
} \
while(0)

HANDLE_GPU_ERROR(0);

// CHECK: #define cbrt(x) pow((double)x, (double)(1.0 / 3.0))
// CHECK-NEXT: double DD = sqrt(cbrt(5.9)) / sqrt(cbrt(3.2));
#define cbrt(x) pow((double)x,(double)(1.0/3.0))
  double DD = sqrt(cbrt(5.9)) / sqrt(cbrt(3.2));

// CHECK: #define NNBI(x) floor(x+0.5)
// CHECK-NEXT: NNBI(3.0);
#define NNBI(x) floor(x+0.5)
NNBI(3.0);

// CHECK: #define PI acos(-1)
#define PI acos(-1)
// CHECK: double cosine = cos(2 * PI);
double cosine = cos(2 * PI);

//CHECK: #define MACRO_KC                                                                    \
//CHECK-NEXT:   q_ct1.parallel_for(                                                          \
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2),   \
//CHECK-NEXT:                           sycl::range<3>(1, 1, 2)),                            \
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { foo_kernel(); });
#define MACRO_KC foo_kernel<<<2, 2, 0>>>();

//CHECK: MACRO_KC
MACRO_KC

//CHECK: #define HARD_KC(NAME, a, b, c, d)                                              \
//CHECK-NEXT:   q_ct1.submit([&](sycl::handler &cgh) {                                       \
//CHECK-NEXT:     int c_ct0 = c;                                                            \
//CHECK-NEXT:     int d_ct1 = d;                                                            \
//CHECK:     cgh.parallel_for(                                                          \
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, a) * sycl::range<3>(1, 1, b),   \
//CHECK-NEXT:                           sycl::range<3>(1, 1, b)),                            \
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { foo3(c_ct0, d_ct1); });               \
//CHECK-NEXT:   });
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1038:{{[0-9]+}}: When the kernel function name is used as a macro argument, the
//CHECK-NEXT: migration result may be incorrect. You need to verify the definition of the
//CHECK-NEXT: macro.
//CHECK-NEXT: */
//CHECK-NEXT: HARD_KC(foo3, 3, 2, 1, 0)
#define HARD_KC(NAME,a,b,c,d) NAME<<<a,b,0>>>(c,d);
HARD_KC(foo3,3,2,1,0)

//CHECK: #define MACRO_KC2(a, b, c, d)                                                       \
//CHECK-NEXT:   q_ct1.submit([&](sycl::handler &cgh) {                                       \
//CHECK-NEXT:     int c_ct0 = c;                                                            \
//CHECK-NEXT:     int d_ct1 = d;                                                            \
//CHECK-NEXT:                                                                                \
//CHECK-NEXT:     cgh.parallel_for(sycl::nd_range<3>(a * b, b),                  \
//CHECK-NEXT:                      [=](sycl::nd_item<3> item_ct1) { foo3(c_ct0, d_ct1); });  \
//CHECK-NEXT:   });
#define MACRO_KC2(a,b,c,d) foo3<<<a, b, 0>>>(c,d);

dim3 griddim = 2;
dim3 threaddim = 32;

// CHECK: MACRO_KC2(griddim,threaddim,1,0)
MACRO_KC2(griddim,threaddim,1,0)

// CHECK: MACRO_KC2(3,2,1,0)
MACRO_KC2(3,2,1,0)

// CHECK: MACRO_KC2(sycl::range<3>(5, 4, 3), 2, 1, 0)
MACRO_KC2(dim3(5,4,3),2,1,0)

int *a;
//CHECK: NESTMACRO3(a = (int *)sycl::malloc_device(100, q_ct1));
NESTMACRO3(hipMalloc(&a,100));

//test if parse error, no check
int b;
#if ( __CUDACC_VER_MAJOR__ >= 8 ) && (__CUDA_ARCH__ >= 600 )
  // DPCT should visit this path
#else
  // If DPCT visit this path, b is redeclared.
  int b;
#endif

  //CHECK: q_ct1.parallel_for(
  //CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 2) * sycl::range<3>(1, 1, 2),
  //CHECK-NEXT:                         sycl::range<3>(1, 1, 2)),
  //CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:         foo_kernel2(3, 3, item_ct1);
  //CHECK-NEXT:       });
  foo_kernel2<<<2, 2, 0>>>(3,3
    #ifdef MACRO_CC
    , 2
    #endif
  );

  #define SIZE3    (100*1024*1024)
  unsigned char *dev_buffer;
  unsigned char *buffer = (unsigned char*)malloc(500);
  //CHECK: q_ct1.memcpy(dev_buffer, buffer, SIZE3).wait();
  hipMemcpy( dev_buffer, buffer, SIZE3, hipMemcpyHostToDevice);
}

// CHECK: template <class T>
// CHECK-NEXT: bool reallocate_host(T **pp, int *curlen, const int newlen,
// CHECK-NEXT:                      /*
// CHECK-NEXT:                      DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not
// CHECK-NEXT:                      meaningful in the migrated code and was removed or replaced
// CHECK-NEXT:                      with 0. You may need to check the migrated code.
// CHECK-NEXT:                      */
// CHECK-NEXT:                      const float fac = 1.0f, const unsigned int flag = 0) {
// CHECK-NEXT:   return true;//reallocate_host_T((void **)pp, curlen, newlen, fac, flag, sizeof(T));
// CHECK-NEXT: }
template <class T>
  bool reallocate_host(T **pp, int *curlen, const int newlen,
                       const float fac=1.0f, const unsigned int flag = hipHostMallocDefault) {
  return true;//reallocate_host_T((void **)pp, curlen, newlen, fac, flag, sizeof(T));
}

bool fooo() {
  int *force_ready_queue;
  int force_ready_queue_size;
  int npatches;
  // CHECK: return reallocate_host<int>(
  // CHECK-NEXT:     &force_ready_queue, &force_ready_queue_size,
  // CHECK-NEXT:     /*
  // CHECK-NEXT:     DPCT1048:{{[0-9]+}}: The original value hipHostMallocMapped is not meaningful in
  // CHECK-NEXT:     the migrated code and was removed or replaced with 0. You may need to
  // CHECK-NEXT:     check the migrated code.
  // CHECK-NEXT:     */
  // CHECK-NEXT:     npatches, 1.2f, 0);
  return reallocate_host<int>(&force_ready_queue, &force_ready_queue_size,
                              npatches, 1.2f, hipHostMallocMapped);
}

void bar() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocDefault is not meaningful in the
  // CHECK-NEXT: migrated code and was removed or replaced with 0. You may need to check the
  // CHECK-NEXT: migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: int i = 0;
  int i = hipHostMallocDefault;
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocMapped is not meaningful in the
  // CHECK-NEXT: migrated code and was removed or replaced with 0. You may need to check the
  // CHECK-NEXT: migrated code.
  // CHECK-NEXT: */
  i = hipHostMallocMapped;
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocPortable is not meaningful in the
  // CHECK-NEXT: migrated code and was removed or replaced with 0. You may need to check the
  // CHECK-NEXT: migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: i = 0;
  i = hipHostMallocPortable;
  // CHECK: /*
  // CHECK-NEXT: DPCT1048:{{[0-9]+}}: The original value hipHostMallocWriteCombined is not meaningful in
  // CHECK-NEXT: the migrated code and was removed or replaced with 0. You may need to check
  // CHECK-NEXT: the migrated code.
  // CHECK-NEXT: */
  // CHECK-NEXT: i = 0;
  i = hipHostMallocWriteCombined;
}
// CHECK: #define BB b
// CHECK-NEXT: #define AAA int *a
// CHECK-NEXT: #define BBB int *BB
#define BB b
#define AAA int *a
#define BBB int *BB

// CHECK: #define CCC AAA, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la, int *b=0
// CHECK-NEXT: #define CC AAA, BBB
#define CCC AAA, int *b=0
#define CC AAA, BBB

// CHECK: #define CCCC(x) void fooc(x)
// CHECK-NEXT: #define CCCCC(x) void foocc(x, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la)
#define CCCC(x) __device__ void fooc(x)
#define CCCCC(x) __device__ void foocc(x)

// CHECK: #define XX(x) void foox(x, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la)
// CHECK-NEXT: #define FF XX(CC)
#define XX(x) __device__ void foox(x)
#define FF XX(CC)

// CHECK: FF
// CHECK-NEXT: {
// CHECK-NEXT: }
FF
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];
}

// CHECK: CCCCC(int *a)
// CHECK-NEXT: {
// CHECK-NEXT: }
CCCCC(int *a)
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];
}


// CHECK: CCCC(CCC)
// CHECK-NEXT: {
// CHECK-NEXT: }
CCCC(CCC)
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];
}

// CHECK: #define FFF void foo(AAA, BBB, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la)
#define FFF __device__ void foo(AAA, BBB)

// CHECK: FFF
// CHECK-NEXT: {
// CHECK-NEXT: }
FFF
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];

}

// CHECK: #define FFFFF(aaa,bbb) void foo4(const int * __restrict__ aaa, const float * __restrict__ bbb, int *c, BBB, const sycl::nd_item<3> &item_ct1, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la)
#define FFFFF(aaa,bbb) __device__ void foo4(const int * __restrict__ aaa, const float * __restrict__ bbb, int *c, BBB)

// CHECK: FFFFF(pos, q)
// CHECK-NEXT: {
// CHECK-EMPTY:
// CHECK-NEXT: const int tid = item_ct1.get_local_id(2);
// CHECK-NEXT: }
FFFFF(pos, q)
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];
  const int tid = threadIdx.x;
}

// CHECK: #define FFFFFF(aaa,bbb) void foo5(const int * __restrict__ aaa, const float * __restrict__ bbb, const sycl::nd_item<3> &item_ct1, float *sp_lj, float *sp_coul, int *ljd, sycl::local_accessor<double, 2> la)
#define FFFFFF(aaa,bbb) __device__ void foo5(const int * __restrict__ aaa, const float * __restrict__ bbb)

// CHECK: FFFFFF(pos, q)
// CHECK-NEXT: {
// CHECK-EMPTY:
// CHECK-NEXT: const int tid = item_ct1.get_local_id(2);
// CHECK-NEXT: }
FFFFFF(pos, q)
{
  __shared__ float sp_lj[4];
  __shared__ float sp_coul[4];
  __shared__ int ljd[1];
  __shared__ double la[8][1];
  const int tid = threadIdx.x;
}

// CHECK: void foo6(AAA, BBB, float *sp_lj, float *sp_coul, int *ljd,
// CHECK-NEXT:   sycl::local_accessor<double, 2> la)
// CHECK-NEXT: {
// CHECK-NEXT: }
__device__ void foo6(AAA, BBB)
{
   __shared__ float sp_lj[4];
   __shared__ float sp_coul[4];
   __shared__ int ljd[1];
   __shared__ double la[8][1];
}


//CHECK: #define MM __umul24
//CHECK-NEXT: #define MUL(a, b) sycl::mul24((unsigned int)a, (unsigned int)b)
//CHECK-NEXT: void foo7(const sycl::nd_item<3> &item_ct1) {
//CHECK-NEXT:   unsigned int tid = MUL(item_ct1.get_local_range(2), item_ct1.get_group(2)) +
//CHECK-NEXT:       item_ct1.get_local_range(2);
//CHECK-NEXT:   unsigned int tid2 = sycl::mul24((unsigned int)item_ct1.get_local_range(2),
//CHECK-NEXT:                                   (unsigned int)item_ct1.get_group_range(2));
//CHECK-NEXT: }
#define MM __umul24
#define MUL(a, b) __umul24(a, b)
__global__ void foo7() {
  unsigned int      tid = MUL(blockDim.x, blockIdx.x) + blockDim.x;
  unsigned int      tid2 = MM(blockDim.x, gridDim.x);
}


//CHECK: void foo8(){
//CHECK-NEXT:   #define SLOW(X) X
//CHECK-NEXT:   double* data;
//CHECK-NEXT:   unsigned long long int tid;
//CHECK-NEXT:   SLOW(dpct::atomic_fetch_add<sycl::access::address_space::generic_space>(
//CHECK-NEXT:            &data[1], tid);
//CHECK-NEXT:        dpct::atomic_fetch_add<sycl::access::address_space::generic_space>(
//CHECK-NEXT:            &data[1], tid + 1);
//CHECK-NEXT:        dpct::atomic_fetch_add<sycl::access::address_space::generic_space>(
//CHECK-NEXT:            &data[2], tid + 2););
//CHECK-NEXT: }
__global__ void foo8(){
#define SLOW(X) X
  double* data;
  unsigned long long int tid;
  SLOW(atomicAdd(&data[1], tid);
  atomicAdd(&data[1], tid + 1);
  atomicAdd(&data[2], tid + 2););
}

//CHECK: #define DFABS(x) (double)sycl::fabs((x))
//CHECK-NEXT: #define MAX(x, y) dpct::max(x, y)
//CHECK-NEXT: void foo9(){
//CHECK-NEXT:   double a,b,c;
//CHECK-NEXT:   MAX(a, sycl::sqrt(DFABS(b)));
//CHECK-NEXT: }
#define DFABS(x) (double) fabs((x))
#define MAX(x, y) max(x, y)
__global__ void foo9(){
  double a,b,c;
  MAX(a, sqrt(DFABS(b)));
}



//CHECK: #define My_PI  3.14159265358979
//CHECK-NEXT: #define g2r(x)  (((double)(x))*My_PI/180)
//CHECK-NEXT: #define sindeg(x) sin(g2r(x))
//CHECK-NEXT: void foo10()
//CHECK-NEXT: {
//CHECK-NEXT:   sindeg(5);
//CHECK-NEXT: }
#define My_PI  3.14159265358979
#define g2r(x)  (((double)(x))*My_PI/180)
#define sindeg(x) sin(g2r(x))
void foo10()
{
  sindeg(5);
}


template<int a, int b>
__global__ void templatefoo(){
  int x = a;
  int y = b;
}
//CHECK: #define AAA 15 + 3
//CHECK-NEXT: #define CCC <<<1,1>>>()
//CHECK-NEXT: #define KERNEL(A, B)                                                           \
//CHECK-NEXT:   dpct::get_in_order_queue().parallel_for(                                      \
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),   \
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { templatefoo<A, B>(); });
//CHECK-NEXT: #define CALL_KERNEL(C, D) KERNEL(C, D); int a = 0;
//CHECK-NEXT: #define CALL_KERNEL2(E, F) CALL_KERNEL(E, F)
//CHECK-NEXT: void templatefoo2(){
//CHECK-NEXT:   CALL_KERNEL2(8, AAA)
//CHECK-NEXT: }
#define AAA 15 + 3
#define CCC <<<1,1>>>()
#define KERNEL(A, B) templatefoo<A,B>CCC
#define CALL_KERNEL(C, D) KERNEL(C, D); int a = 0;
#define CALL_KERNEL2(E, F) CALL_KERNEL(E, F)
void templatefoo2(){
  CALL_KERNEL2(8, AAA)
}

//CHECK: void foo11(const sycl::nd_item<3> &item_ct1){
//CHECK-NEXT:   sycl::exp((double)(THREAD_IDX_X));
//CHECK-NEXT: }
__global__ void foo11(){
  exp(THREAD_IDX_X);
}

//CHECK: /*
//CHECK-NEXT: DPCT1055:{{[0-9]+}}: Vector types with size 1 are migrated to the corresponding
//CHECK-NEXT: fundamental types, which cannot be inherited. You need to rewrite the code.
//CHECK-NEXT: */
//CHECK-NEXT: #define VECTOR_TYPE_DEF(type)                                                  \
//CHECK-NEXT:                                                                                \
//CHECK-NEXT:   struct MyVector : type                                                       \
//CHECK-NEXT:   {                                                                            \
//CHECK-NEXT:     typedef type Type;                                                         \
//CHECK-NEXT:     __dpct_inline__ MyVector operator+(const MyVector &other) const {          \
//CHECK-NEXT:       MyVector retval;                                                         \
//CHECK-NEXT:       retval = *this + other;                                                  \
//CHECK-NEXT:       return retval;                                                           \
//CHECK-NEXT:     }                                                                          \
//CHECK-NEXT:     __dpct_inline__ MyVector operator-(const MyVector &other) const {          \
//CHECK-NEXT:       MyVector retval;                                                         \
//CHECK-NEXT:       retval = *this - other;                                                  \
//CHECK-NEXT:       return retval;                                                           \
//CHECK-NEXT:     }                                                                          \
//CHECK-NEXT:   };                                                                           \
//CHECK-NEXT:                                                                                \
//CHECK-NEXT:   struct MyVector2 : sycl::type##2                                             \
//CHECK-NEXT:   {                                                                            \
//CHECK-NEXT:     typedef sycl::type##2 Type;                                                \
//CHECK-NEXT:     __dpct_inline__ MyVector2 operator+(const MyVector2 &other) const {        \
//CHECK-NEXT:       MyVector2 retval;                                                        \
//CHECK-NEXT:       retval.x() = x() + other.x();                                            \
//CHECK-NEXT:       retval.y() = y() + other.y();                                            \
//CHECK-NEXT:       return retval;                                                           \
//CHECK-NEXT:     }                                                                          \
//CHECK-NEXT:     __dpct_inline__ MyVector2 operator-(const MyVector2 &other) const {        \
//CHECK-NEXT:       MyVector2 retval;                                                        \
//CHECK-NEXT:       retval.x() = x() - other.x();                                            \
//CHECK-NEXT:       retval.y() = y() - other.y();                                            \
//CHECK-NEXT:       return retval;                                                           \
//CHECK-NEXT:     }                                                                          \
//CHECK-NEXT:   };

#define VECTOR_TYPE_DEF(type)                                                                           \
                                                                                                        \
    struct MyVector : type##1                                                                           \
    {                                                                                                   \
        typedef type##1   Type;                                                                         \
        __host__ __device__ __forceinline__ MyVector operator+(const MyVector &other) const {           \
        MyVector retval;                                                                                \
            retval.x = x + other.x;                                                                     \
            return retval;                                                                              \
        }                                                                                               \
        __host__ __device__ __forceinline__ MyVector operator-(const MyVector &other) const {           \
        MyVector retval;                                                                                \
            retval.x = x - other.x;                                                                     \
            return retval;                                                                              \
        }                                                                                               \
    };                                                                                                  \
                                                                                                        \
    struct MyVector2 : type##2                                                                          \
    {                                                                                                   \
        typedef type##2 Type;                                                                           \
        __host__ __device__ __forceinline__ MyVector2 operator+(const MyVector2 &other) const {         \
            MyVector2 retval;                                                                           \
            retval.x = x + other.x;                                                                     \
            retval.y = y + other.y;                                                                     \
            return retval;                                                                              \
        }                                                                                               \
        __host__ __device__ __forceinline__ MyVector2 operator-(const MyVector2 &other) const {         \
            MyVector2 retval;                                                                           \
            retval.x = x - other.x;                                                                     \
            retval.y = y - other.y;                                                                     \
            return retval;                                                                              \
        }                                                                                               \
    };

VECTOR_TYPE_DEF(int)

//CHECK: typedef float real;
//CHECK-NEXT: #define POW(x, y) dpct::pow(x, y)
//CHECK-NEXT: #define POW2(x, y) x *x
//CHECK-NEXT: /*
//CHECK-NEXT: DPCT1064:{{[0-9]+}}: Migrated pow call is used in a macro/template definition and may
//CHECK-NEXT: not be valid for all macro/template uses. Adjust the code.
//CHECK-NEXT: */
//CHECK-NEXT: #define POW3(x, y) dpct::pow(x, y)
//CHECK: #define SQRT(x) sycl::sqrt(x)
//CHECK-NEXT: void foo12(){
//CHECK-NEXT: real *vx;
//CHECK-NEXT: real *vy;
//CHECK-NEXT: int id;
//CHECK-NEXT: real v2 = SQRT(SQRT(POW(vx[id], 2.0) + POW(vy[id], 2.0)));
//CHECK-NEXT: real v3 = POW2(vx[id], 2);
//CHECK-NEXT: real v4 = POW3(vx[id], 3.0);
//CHECK-NEXT: real v5 = POW3(vx[id], 2);
//CHECK-NEXT: }
typedef float real;
#define POW(x,y)    powf(x,y)
#define POW2(x,y)    pow(x,y)
#define POW3(x,y)    pow(x,y)
#define SQRT(x)     sqrtf(x)
__global__ void foo12(){
real *vx;
real *vy;
int id;
real v2 = SQRT(SQRT(POW(vx[id], 2.0) + POW(vy[id], 2.0)));
real v3 = POW2(vx[id], 2);
real v4 = POW3(vx[id], 3.0);
real v5 = POW3(vx[id], 2);
}

//CHECK: #define CALL(call) call;
//CHECK-NEXT: #define SIZE2 8
//CHECK-NEXT: void foo13(){
//CHECK-NEXT:   int *a;
//CHECK-NEXT:   CALL(a = sycl::malloc_device<int>(SIZE2 * 10, dpct::get_in_order_queue()));
//CHECK-NEXT: }
#define CALL(call) call;
#define SIZE2 8
void foo13(){
  int *a;
  CALL(hipMalloc(&a, SIZE2 * 10 * sizeof(int)));
}

//CHECK: #define CONST const
//CHECK-NEXT: #define INT2 sycl::int2
//CHECK-NEXT: #define PTR *
//CHECK-NEXT: #define PTR2 PTR
//CHECK-NEXT: #define ALL const sycl::int2 *
//CHECK-NEXT: #define TYPE_PTR(T) T *
//CHECK-NEXT: #define ALL2(C, T, P) C T P
//CHECK-NEXT: #define ALL3(X) X
#define CONST const
#define INT2 int2
#define PTR *
#define PTR2 PTR
#define ALL const int2 *
#define TYPE_PTR(T) T *
#define ALL2(C, T, P) C T P
#define ALL3(X) X

//CHECK: int foo14(){
//CHECK-NEXT:   const sycl::int2 *aaa;
//CHECK-NEXT:   CONST sycl::int3 *bbb;
//CHECK-NEXT:   ALL3(const sycl::int2 *) ccc;
//CHECK-NEXT:   ALL2(const, sycl::int2, *) ddd;
//CHECK-NEXT:   ALL3(const) ALL3(sycl::int2) ALL3(*) eee;
//CHECK-NEXT:   ALL fff;
//CHECK-NEXT:   CONST INT2 PTR ggg;
//CHECK-NEXT:   CONST INT2 PTR2 hhh;
//CHECK-NEXT:   CONST sycl::int3 PTR2 iii;
//CHECK-NEXT:   TYPE_PTR(sycl::int2) jjj;
//CHECK-NEXT:   ALL3(ALL3(const sycl::int2 *)) kkk;
//CHECK-NEXT:   ALL2(const, ALL3(sycl::int2), *) lll;
//CHECK-NEXT: }
int foo14(){
  const int2 *aaa;
  CONST int3 *bbb;
  ALL3(const int2 *) ccc;
  ALL2(const, int2, *) ddd;
  ALL3(const) ALL3(int2) ALL3(*) eee;
  ALL fff;
  CONST INT2 PTR ggg;
  CONST INT2 PTR2 hhh;
  CONST int3 PTR2 iii;
  TYPE_PTR(int2) jjj;
  ALL3(ALL3(const int2 *)) kkk;
  ALL2(const, ALL3(int2), *) lll;
}

//CHECK: #define FABS(a) (sycl::fabs((float)((a).x())) + sycl::fabs((float)((a).y())))
//CHECK-NEXT: static inline double foo16(const sycl::float2 &x) { return FABS(x); }
#define FABS(a)       (fabs((a).x) + fabs((a).y))
__host__ __device__ static inline double foo16(const float2 &x) { return FABS(x); }

//CHECK: #define _mulhilo_(W, Word, NAME)                                               \
//CHECK-NEXT: Word mulhilo##W(Word a, Word b, Word *hip) {                                 \
//CHECK-NEXT:     *hip = NAME(a, b);                                                         \
//CHECK-NEXT:     return a * b;                                                              \
//CHECK-NEXT: }
//CHECK-NEXT: _mulhilo_(64, uint64_t, sycl::mul_hi)
#include "hip/hip_fp16.h"
#define _mulhilo_(W, Word, NAME)                       \
__device__ Word mulhilo##W(Word a, Word b, Word* hip) { \
    *hip = NAME(a, b);                                 \
    return a*b;                                        \
}
_mulhilo_(64, uint64_t, __umul64hi)




//CHECK: #define AAA __heq
//CHECK-NEXT: #define CALL(x) x
//CHECK-NEXT: #define CALL2(x) CALL(x)
//CHECK-NEXT: #define III CALL(CALL(CALL(h == h_1)))
//CHECK-NEXT: #define JJJ CALL(CALL(CALL(III)))
//CHECK-NEXT: #define KKK JJJ
//CHECK-NEXT: void foo16() {
//CHECK-NEXT:     sycl::half h, h_1, h_2;
//CHECK-NEXT:     sycl::half2 h2, h2_1, h2_2;
//CHECK-NEXT:     bool b;
//CHECK-NEXT:     CALL(CALL(CALL(JJJ)));
//CHECK-NEXT: }
#define AAA __heq
#define CALL(x) x
#define CALL2(x) CALL(x)
#define III CALL(CALL(CALL(AAA (h, h_1))))
#define JJJ CALL(CALL(CALL(III)))
#define KKK JJJ
__global__ void foo16() {
    __half h, h_1, h_2;
    __half2 h2, h2_1, h2_2;
    bool b;
    CALL(CALL(CALL(JJJ)));
}

// [Todo] Macro issue here will fix in issue jira
void foo17(){
  size_t result1, result2;
  int size = 32;
  float* f_A;
  // Error CALL() will be removed
  CALL(HIP_MEMCPY3D cpy2);
  hipDeviceptr_t f_D = 0;
  CALL(hipMalloc(&f_D, size));
}

//CHECK: #define CONCATE(name) cuda##name
//CHECK-NEXT: typedef dpct::queue_ptr stream_t2;
//CHECK-NEXT: typedef dpct::event_ptr event_t2;
#define CONCATE(name) cuda##name
typedef CONCATE(Stream_t) stream_t2;
typedef CONCATE(Event_t) event_t2;

//CHECK: void foo18() {
//CHECK-NEXT:   dpct::device_ext &dev_ct1 = dpct::get_current_device();
//CHECK-NEXT:   dpct::event_ptr event;
//CHECK-NEXT:   event->wait_and_throw();
//CHECK-NEXT:   stream_t2 *stream;
//CHECK-NEXT:   stream_t2 stream2;
//CHECK-NEXT:   *(stream) = dev_ct1.create_queue();
//CHECK-NEXT:   unsigned int flags;
//CHECK-NEXT:   /*
//CHECK-NEXT:   DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag and priority options.
//CHECK-NEXT:   */
//CHECK-NEXT:   *(stream) = dev_ct1.create_queue();
//CHECK-NEXT:   int priority;
//CHECK-NEXT:   /*
//CHECK-NEXT:   DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag and priority options.
//CHECK-NEXT:   */
//CHECK-NEXT:   *(stream) = dev_ct1.create_queue();
//CHECK-NEXT:   dev_ct1.destroy_queue(stream2);
//CHECK-NEXT: }
void foo18(){
  hipEvent_t event;
  CONCATE(EventSynchronize)(event);
  stream_t2 *stream;
  stream_t2 stream2;
  CONCATE(StreamCreate)(stream);
  unsigned int flags;
  CONCATE(StreamCreateWithFlags)(stream, flags);
  int priority;
  CONCATE(StreamCreateWithPriority)(stream, flags, priority);
  CONCATE(StreamDestroy)(stream2);
}

// CHECK: static const int streamDefault2 = 0;
// CHECK-NEXT: static const int streamDefault = CALL(0);
// CHECK-NEXT: static const int streamNonBlocking = 0;
// CHECK-NEXT: static const dpct::queue_ptr streamDefault3 = &dpct::get_in_order_queue();
// CHECK-NEXT: static const dpct::queue_ptr streamDefault4 = CALL(&dpct::get_in_order_queue());
static const int streamDefault2 = hipStreamDefault;
static const int streamDefault = CALL(CONCATE(StreamDefault));
static const int streamNonBlocking = CONCATE(StreamNonBlocking);
static const hipStream_t streamDefault3 = hipStreamDefault;
static const hipStream_t streamDefault4 = CALL(hipStreamDefault);

//     CHECK:#define CMC_PROFILING_BEGIN()                                                  \
//CHECK-NEXT:  dpct::event_ptr start;                                                       \
//CHECK-NEXT:  dpct::event_ptr stop;                                                        \
//CHECK-NEXT:  if (CMC_profile)                                                             \
//CHECK-NEXT:  {                                                                            \
//CHECK-NEXT:    start = new sycl::event();                                                 \
//CHECK-NEXT:    stop = new sycl::event();                                                  \
//CHECK-NEXT:    *start = q_ct1.ext_oneapi_submit_barrier();                                \
//CHECK-NEXT:  }
#define CMC_PROFILING_BEGIN()                                                                                      \
  hipEvent_t start;                                                                                               \
  hipEvent_t stop;                                                                                                \
  if (CMC_profile)                                                                                                 \
  {                                                                                                                \
    hipEventCreate(&start);                                                                                       \
    hipEventCreate(&stop);                                                                                        \
    hipGetLastError();                                                                                            \
    hipEventRecord(start);                                                                                        \
  }

//     CHECK:#define CMC_PROFILING_END(lineno)                                              \
//CHECK-NEXT:  if (CMC_profile)                                                             \
//CHECK-NEXT:  {                                                                            \
//CHECK-NEXT:    *stop = q_ct1.ext_oneapi_submit_barrier();                                 \
//CHECK-NEXT:    stop->wait_and_throw();                                                    \
//CHECK-NEXT:    float time = 0.0f;                                                         \
//CHECK-NEXT:    time = (stop->get_profiling_info<                                          \
//CHECK-NEXT:                sycl::info::event_profiling::command_end>() -                  \
//CHECK-NEXT:            start->get_profiling_info<                                         \
//CHECK-NEXT:                sycl::info::event_profiling::command_start>()) /               \
//CHECK-NEXT:           1000000.0f;                                                         \
//CHECK-NEXT:    dpct::destroy_event(start);                                                \
//CHECK-NEXT:    dpct::destroy_event(stop);                                                 \
//CHECK-NEXT:  }                                                                            \
//CHECK-NEXT:  dpct::err0 error = 0;
#define CMC_PROFILING_END(lineno)                                                                          \
  if (CMC_profile)                                                                                         \
  {                                                                                                        \
    hipEventRecord(stop);                                                                                 \
    hipEventSynchronize(stop);                                                                            \
    float time = 0.0f;                                                                                     \
    hipEventElapsedTime(&time, start, stop);                                                              \
    hipEventDestroy(start);                                                                               \
    hipEventDestroy(stop);                                                                                \
  }                                                                                                        \
  hipError_t error = hipGetLastError();                                                                  \
  if (error)                                                                                               \
  {                                                                                                        \
    printf("%s\nCUDA ERROR!!! Detected at end of CMC_PROFILING_END in BsplineJastrowCudaPBC line %d!!!\n", \
           hipGetErrorString(error),                                                                      \
           lineno);                                                                                        \
    exit(1);                                                                                               \
  }

void foo20() {
  bool CMC_profile = true;
  CMC_PROFILING_BEGIN();
  CMC_PROFILING_END(__LINE__);
}

//CHECK: #define CALLSHFLSYNC(x)                                                        \
//CHECK-NEXT: dpct::select_from_sub_group(item_ct1.get_sub_group(), x, 3 ^ 1);
#define CALLSHFLSYNC(x) __shfl_sync(0xffffffff, x, 3 ^ 1);
//CHECK: #define CALLANYSYNC(x)                                                         \
//CHECK-NEXT:   sycl::any_of_group(                                                          \
//CHECK-NEXT:       item_ct1.get_sub_group(),                                                \
//CHECK-NEXT:       (0xffffffff &                                                            \
//CHECK-NEXT:        (0x1 << item_ct1.get_sub_group().get_local_linear_id())) &&             \
//CHECK-NEXT:           x != 0.0f);
#define CALLANYSYNC(x) __any_sync(0xffffffff, x != 0.0f);

__global__ void foo21(){
  int a;
  CALLSHFLSYNC(a);
  CALLANYSYNC(a);
}


//CHECK: #define FUNCNAME(x) x
//CHECK-NEXT: #define PASS(x) x
//CHECK-NEXT: template <typename T, int X, int Y>
//CHECK-NEXT: void doo(float f, const sycl::stream &stream_ct1) {
//CHECK-NEXT:   stream_ct1 << "doo\n";
//CHECK-NEXT: }
#define FUNCNAME(x) x
#define PASS(x) x
template <typename T, int X, int Y>
__device__ void doo(float f) {
  printf("doo\n");
}

//CHECK: void foo22(const sycl::stream &stream_ct1) {
//CHECK-NEXT:   FUNCNAME(doo)<float, PASS(1 +) 2, SIZE2>(PASS(1 +) 0.0f, stream_ct1);
//CHECK-NEXT: }
__global__ void foo22() {
  FUNCNAME(doo)<float, PASS(1 +) 2, SIZE2>(PASS(1 +) 0.0f);
}

//CHECK: static __inline__ void __attribute__((__always_inline__, __nodebug__, __target__("mmx")))
//CHECK-NEXT: foo23(void)
//CHECK-NEXT: {
//CHECK-NEXT:     __builtin_ia32_emms();
//CHECK-NEXT: }
static __inline__ void __attribute__((__always_inline__, __nodebug__, __target__("mmx")))
foo23(void)
{
  __builtin_ia32_emms();
}

//CHECK: #define SHFL(x, y, z)                                                          \
//CHECK-NEXT: dpct::select_from_sub_group(item_ct1.get_sub_group(), (x), (y), (z))
#define SHFL(x, y, z) __shfl((x), (y), (z))
__global__ void foo24(){
  int i;
  SHFL(i, i, 16);
}


#include <hipblas.h>
int foo25(){
//CHECK: #if defined(MKL_SYCL_HPP)
#if defined(CUBLAS_V2_H_)
#endif

//CHECK: #ifndef MKL_SYCL_HPP
//CHECK-NEXT: #define CUBLAS_V2_H_
#ifndef CUBLAS_V2_H_
#define CUBLAS_V2_H_
float *h_a, *h_b, *h_c;
float *d_C_S;
int a = hipblasSetVector(10, sizeof(float), h_a, 11111, d_C_S, 11111);
#endif
return 0;
}

//CHECK:#define AAAAA_Z_MAKE(r, i) sycl::double2(r, i)
//CHECK-NEXT:#define AAAAA_Z_ZERO AAAAA_Z_MAKE(0.0, 0.0)
//CHECK-NEXT:void aaaaa_zprint_vector() {
//CHECK-NEXT:  sycl::double2 z_zero = AAAAA_Z_ZERO;
//CHECK-NEXT:#ifdef COMPLEX
//CHECK-NEXT:#define AAA
//CHECK-NEXT:#else
//CHECK-NEXT:#define BBB
//CHECK-NEXT:#endif
//CHECK-NEXT:}
#define AAAAA_Z_MAKE(r, i) make_hipDoubleComplex(r, i)
#define AAAAA_Z_ZERO AAAAA_Z_MAKE(0.0, 0.0)
void aaaaa_zprint_vector() {
  hipDoubleComplex z_zero = AAAAA_Z_ZERO;
#ifdef COMPLEX
#define AAA
#else
#define BBB
#endif
}

namespace launch_bounds_test {
constexpr uint32_t AAAAA_launch_bounds_test = 1024;
constexpr uint32_t BBBBB_launch_bounds_test = 256;
#define CCCCC_launch_bounds_test(val)          \
  (((val) <= AAAAA_launch_bounds_test) ? (val) \
      : BBBBB_launch_bounds_test)

// CHECK: #define DDDDD_launch_bounds_test(max_threads_per_block) \
// CHECK-NEXT: /*comment*/
// CHECK-NEXT: template <typename T1, typename T2, int I>
#define DDDDD_launch_bounds_test(max_threads_per_block) \
  __launch_bounds__((CCCCC_launch_bounds_test((max_threads_per_block)))) /*comment*/
template <typename T1, typename T2, int I>
DDDDD_launch_bounds_test(512)
__global__ void test() {}

// CHECK: #define EEEEE_launch_bounds_test(max_threads_per_block) \
// CHECK-EMPTY:
// CHECK-NEXT: template <typename T1, typename T2, int I>
#define EEEEE_launch_bounds_test(max_threads_per_block) \
__launch_bounds__((CCCCC_launch_bounds_test((max_threads_per_block))))
template <typename T1, typename T2, int I>
EEEEE_launch_bounds_test(512)
__global__ void test2() {}

#undef CCCCC_launch_bounds_test
#undef DDDDD_launch_bounds_test
#undef EEEEE_launch_bounds_test
}

//     CHECK:#if (defined(DPCT_COMPATIBILITY_TEMP) &&                                       \
//CHECK-NEXT:     !(defined(__clang__) && defined(SYCL_LANGUAGE_VERSION)))
//CHECK-NEXT:__host__ __device__
//CHECK-NEXT:#endif
//CHECK-NEXT:void foo26 () {}
#if (defined(__CUDA_ARCH__) && !(defined(__clang__) && defined(__CUDA__)))
__host__ __device__
#endif
void foo26 () {}

// check not to assert
//CHECK: namespace user_namespace {
//CHECK-NEXT:   template <typename T> struct hipfftDoubleComplex {};
//CHECK-NEXT: }
//CHECK-NEXT: #define MACRO_AA(ARG) ARG()
//CHECK-NEXT: template <typename T> void bar() {}
//CHECK-NEXT: #define MACRO_BB() void foo27() { return bar<user_namespace::hipfftDoubleComplex<float>>(); }
//CHECK-NEXT: MACRO_AA(MACRO_BB)
namespace user_namespace {
  template <typename T> struct hipfftDoubleComplex {};
}
#define MACRO_AA(ARG) ARG()
template <typename T> void bar() {}
#define MACRO_BB() void foo27() { return bar<user_namespace::hipfftDoubleComplex<float>>(); }
MACRO_AA(MACRO_BB)


#define CALL_K(...) __VA_ARGS__
void foo28(){
  //CHECK: CALL_K(dpct::get_in_order_queue().parallel_for(
  //CHECK-NEXT:   sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  //CHECK-NEXT:   [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:     foo_kernel();
  //CHECK-NEXT:   });)
  CALL_K(foo_kernel<<<1,1,0>>>();)
}



#define SIMD_SIZE 32
#define BLOCK_PAIR 256

#define local_allocate_store_charge()                                       \
    __shared__ double red_acc[8][BLOCK_PAIR / SIMD_SIZE];

//CHECK: void foo29(sycl::local_accessor<double, 2> red_acc) {
//CHECK-NEXT: }
__global__ void foo29() {
  local_allocate_store_charge();
}

template<class T1, class T2, int N> __global__ void foo31();

//CHECK: #define FOO31(DIMS)                                                            \
//CHECK-NEXT: q_ct1.parallel_for(                                                          \
//CHECK-NEXT:     sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),     \
//CHECK-NEXT:     [=](sycl::nd_item<3> item_ct1) { foo31<unsigned int, float, DIMS>(); });


#define FOO31(DIMS) foo31<unsigned int, float, DIMS><<<1,1>>>();

//CHECK: {
//CHECK-NEXT:   dpct::has_capability_or_fail(q_ct1.get_device(), {sycl::aspect::fp64});
//CHECK-EMPTY:
//CHECK-NEXT:   q_ct1.submit([&](sycl::handler &cgh) {
//CHECK-NEXT:     /*
//CHECK-NEXT:     DPCT1101:{{[0-9]+}}: 'BLOCK_PAIR / SIMD_SIZE' expression was replaced with a
//CHECK-NEXT:     value. Modify the code to use the original expression, provided in
//CHECK-NEXT:     comments, if it is correct.
//CHECK-NEXT:     */
//CHECK-NEXT:     sycl::local_accessor<double, 2> red_acc_acc_ct1(
//CHECK-NEXT:         sycl::range<2>(8, 8 /*BLOCK_PAIR / SIMD_SIZE*/), cgh);

//CHECK:     cgh.parallel_for(
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
//CHECK-NEXT:           foo29(red_acc_acc_ct1);
//CHECK-NEXT:         });
//CHECK-NEXT:   });
//CHECK-NEXT:   }
//CHECK-NEXT:   FOO31(1)
//CHECK-NEXT: }
void foo30(){
  foo29<<<1,1,0>>>();
  FOO31(1)
}



#define VA_CALL2(...) __VA_ARGS__()
#define VA_CALL(...) VA_CALL2(__VA_ARGS__)

template<class T>
__global__ void template_kernel(T t){
    __shared__ T t2;
}

int foo31(){
  //CHECK: VA_CALL(([&] {
  //CHECK-NEXT:   dpct::get_in_order_queue().submit([&](sycl::handler &cgh) {
  //CHECK-NEXT:     sycl::local_accessor<int, 0> t2_acc_ct1(cgh);
  //CHECK:     cgh.parallel_for(
  //CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),
  //CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) {
  //CHECK-NEXT:           template_kernel<int>(10, t2_acc_ct1);
  //CHECK-NEXT:         });
  //CHECK-NEXT:   });
  //CHECK-NEXT: }));
  VA_CALL( ([&]{ template_kernel<int><<<1,1,0>>>(10); }) );
}

class ArgClass{};

//CHECK: #define SIZE 256
#define SIZE 256
//CHECK: #define VACALL4(...) __VA_ARGS__()
//CHECK-NEXT: #define VACALL3(...) VACALL4(__VA_ARGS__)
//CHECK-NEXT: #define VACALL2(...) VACALL3(__VA_ARGS__)
//CHECK-NEXT: #define VACALL(x)                                                              \
//CHECK-NEXT:   dpct::get_in_order_queue().submit([&](sycl::handler &cgh) {                   \
//CHECK-NEXT:     int i_ct0 = i;                                                            \
//CHECK-NEXT:     auto ac_ct0 = ac;                                                          \
//CHECK:     cgh.parallel_for(                                                          \
//CHECK-NEXT:         sycl::nd_range<3>(sycl::range<3>(1, 1, 2) *                            \
//CHECK-NEXT:                               sycl::range<3>(1, 1, SIZE),                      \
//CHECK-NEXT:                           sycl::range<3>(1, 1, SIZE)),                         \
//CHECK-NEXT:         [=](sycl::nd_item<3> item_ct1) { foo32(i_ct0, ac_ct0); });             \
//CHECK-NEXT:   });
#define VACALL4(...) __VA_ARGS__()
#define VACALL3(...) VACALL4(__VA_ARGS__)
#define VACALL2(...) VACALL3(__VA_ARGS__)
#define VACALL(x) foo32<<<2,SIZE,0>>>(i, ac)
__global__ void foo32(int a, ArgClass ac){}

// CHECK: int foo33(){
// CHECK-NEXT:   ArgClass ac;
// CHECK-NEXT:   int i;
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1038:{{[0-9]+}}: When the kernel function name is used as a macro argument, the
// CHECK-NEXT:   migration result may be incorrect. You need to verify the definition of the
// CHECK-NEXT:   macro.
// CHECK-NEXT:   */
// CHECK-NEXT:   VACALL2([&] {VACALL(0);
// CHECK-NEXT:   });
// CHECK-NEXT: }
int foo33(){
  ArgClass ac;
  int i;
  VACALL2([&]{VACALL(0);});
}


#include <thrust/device_vector.h>
#include <thrust/host_vector.h>
#include <thrust/unique.h>

void foo34() {

  int *ptr;
  thrust::host_vector<int> h_keys, h_values;
  thrust::device_vector<int> d_keys, d_values;
  thrust::equal_to<int> binary_pred;

  auto dummy_dev = thrust::device_ptr<int>(ptr);
  int numel = 1;
  using index_t = int;
  VACALL3([&]() {
    int64_t num_of_segments;
    {
      auto sorted_indices_dev = thrust::device_ptr<index_t>(ptr);
      auto dummy_dev = thrust::device_ptr<index_t>(ptr);
// CHECK:      auto ends =
// CHECK-NEXT: dpct::unique_copy(oneapi::dpl::execution::make_device_policy(
// CHECK-NEXT:                       dpct::get_in_order_queue()),
// CHECK-NEXT:                   sorted_indices_dev, sorted_indices_dev + numel,
// CHECK-NEXT:                   dpct::make_counting_iterator(0), dummy_dev,
// CHECK-NEXT:                   dpct::device_pointer<index_t>(ptr));
      auto ends = thrust::unique_by_key_copy(
          thrust::device, sorted_indices_dev, sorted_indices_dev + numel,
          thrust::make_counting_iterator(0), dummy_dev,
          thrust::device_ptr<index_t>(ptr));
    }
  });
}


//CHECK: #define ReturnErrorFunction                                                    \
//CHECK-NEXT:   int amax(dpct::blas::descriptor_ptr handle, const int n, const float *X,     \
//CHECK-NEXT:            const int incX, int &result) try {                                  \
//CHECK-NEXT:     return [&]() {                                                             \
//CHECK-NEXT:       dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(                    \
//CHECK-NEXT:           handle->get_queue(), &result);                                       \
//CHECK-NEXT:       oneapi::mkl::blas::column_major::iamax(handle->get_queue(), n, X, incX,  \
//CHECK-NEXT:                                              res_wrapper_ct4.get_ptr(),        \
//CHECK-NEXT:                                              oneapi::mkl::index_base::one);    \
//CHECK-NEXT:       return 0;                                                                \
//CHECK-NEXT:     }();                                                                       \
//CHECK-NEXT:   }                                                                            \
//CHECK-NEXT:   catch (sycl::exception const &exc) {                                         \
//CHECK-NEXT:     std::cerr << exc.what() << "Exception caught at file:" << __FILE__         \
//CHECK-NEXT:               << ", line:" << __LINE__ << std::endl;                           \
//CHECK-NEXT:     std::exit(1);                                                              \
//CHECK-NEXT:   }

#define ReturnErrorFunction                                                         \
  hipblasStatus_t amax( hipblasHandle_t handle,                                       \
                       const int n, const float* X, const int incX, int& result )   \
  {                                                                                 \
    return hipblasIsamax(handle, n, (const float*) X, incX, &result);                \
  }

ReturnErrorFunction

#define CUSOLVER_CHECK(err)                                                    \
  do {                                                                         \
    hipsolverStatus_t err_ = (err);                                             \
    if (err_ != HIPSOLVER_STATUS_SUCCESS) {                                     \
      printf("cusolver error %d at %s:%d\n", err_, __FILE__, __LINE__);        \
      throw std::runtime_error("cusolver error");                              \
    }                                                                          \
  } while (0)

void foo35() {
  hipsolverHandle_t handle;
  const int m = 3;
  double *d_A;
  const int lda = m;
  int lwork = 0;
  //CHECK: CUSOLVER_CHECK(DPCT_CHECK_ERROR(
  //CHECK-NEXT:   lwork = oneapi::mkl::lapack::geqrf_scratchpad_size<double>(*handle, m, m,
  //CHECK-NEXT:                                                              lda)));
  CUSOLVER_CHECK(hipsolverDnDgeqrf_bufferSize(handle, m, m, d_A, lda, &lwork));
}

#undef CUSOLVER_CHECK

template<class T>
class TemplateClass{};

template<class a>
__global__ void templatefoo3(){}

//CHECK: #define CALLTEMPLATEFOO                                                        \
//CHECK-NEXT:   q_ct1.parallel_for(                                                          \
//CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),     \
//CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) {                                         \
//CHECK-NEXT:         templatefoo3<TemplateClass<TemplateClass<int>>>();                     \
//CHECK-NEXT:       });
//CHECK-NEXT: #define CALLTEMPLATEFOO2                                                       \
//CHECK-NEXT:   q_ct1.parallel_for(                                                          \
//CHECK-NEXT:       sycl::nd_range<3>(sycl::range<3>(1, 1, 1), sycl::range<3>(1, 1, 1)),     \
//CHECK-NEXT:       [=](sycl::nd_item<3> item_ct1) { templatefoo3<TemplateClass<int>>(); });

#define CALLTEMPLATEFOO templatefoo3<TemplateClass<TemplateClass<int>>><<<1,1,0>>>()
#define CALLTEMPLATEFOO2 templatefoo3<TemplateClass<int>><<<1,1,0>>>()
void foo36() {
  CALLTEMPLATEFOO;
  CALLTEMPLATEFOO2;
}

template<typename T>void foo37(const T* t){}
#define FOO37(T)  template void foo37(const T* t)
//CHECK: FOO37(sycl::half);
FOO37(half);
#endif


