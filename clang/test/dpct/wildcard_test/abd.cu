
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --usm-level=none -out-root=%T/abd -in-root=%S %S/ab*.cu --cuda-include-path="%cuda-path/include" --sycl-named-lambda -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/abd/abd.dp.cpp --match-full-lines %S/abd.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/abd/abd.dp.cpp -o %T/abd/abd.dp.o %}
// RUN: FileCheck --input-file %T/abd/abc.dp.cpp --match-full-lines %S/abc.cu
// RUN: %if build_lit %{icpx -c -fsycl %T/abd/abc.dp.cpp -o %T/abd/abc.dp.o %}

int printf(const char *format, ...);

const char *test_function() {

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced by a placeholder string. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "<Placeholder string>");
  printf("%s\n", hipGetErrorString(hipGetLastError()));
}
