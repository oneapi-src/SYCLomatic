#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none -out-root %T/replace-cuda-header-includes %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/replace-cuda-header-includes/replace-cuda-header-includes.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/replace-cuda-header-includes/replace-cuda-header-includes.dp.cpp -o %T/replace-cuda-header-includes/replace-cuda-header-includes.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include "test-header.dp.hpp"
// CHECK: // First function
#include "test-header.cuh"
#include <hip/hip_runtime.h>
#include <hip/device_functions.h>
#include <hipfft/hipfft.h>
// First function
__global__ void foo() {
  // CHECK: size_t tix = item_ct1.get_local_id(2);
  // CHECK: size_t tiy = item_ct1.get_local_id(1);
  // CHECK: size_t tiz = item_ct1.get_local_id(0);

  size_t tix = threadIdx.x;
  size_t tiy = threadIdx.y;
  size_t tiz = threadIdx.z;

  // size_t bix = blockIdx.x;
  // size_t biy = blockIdx.y;
  // size_t biz = blockIdx.z;

  // CHECK: size_t bdx = item_ct1.get_local_range(2);
  // CHECK: size_t bdy = item_ct1.get_local_range(1);
  // CHECK: size_t bdz = item_ct1.get_local_range(0);

  size_t bdx = blockDim.x;
  size_t bdy = blockDim.y;
  size_t bdz = blockDim.z;

  // size_t gdx = gridDim.x;
  // size_t gdy = gridDim.y;
  // size_t gdz = gridDim.z;
}

