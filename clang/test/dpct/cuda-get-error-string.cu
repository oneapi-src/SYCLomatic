
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none -out-root %T/cuda-get-error-string %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/cuda-get-error-string/cuda-get-error-string.dp.cpp

int printf(const char *format, ...);

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR(X) printf("%s\n", "cudaGetErrorString is not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR(X) printf("%s\n", hipGetErrorString(X))

// CHECK:  /*
// CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT: #define PRINT_ERROR_STR2(X)\
// CHECK-NEXT:  printf("%s\n", "cudaGetErrorString is not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR2(X)\
  printf("%s\n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", "cudaGetErrorString is not supported"/*cudaGetErrorString(X)*/)
#define PRINT_ERROR_STR3(X)\
  printf("%s\
         \n", hipGetErrorString(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME(X) printf("%s\n", "cudaGetErrorName is not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME(X) printf("%s\n", hipGetErrorName(X))

// CHECK:   /*
// CHECK-NEXT:   DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT:   */
// CHECK-NEXT: #define PRINT_ERROR_NAME2(X)\
// CHECK-NEXT:   printf("%s\n", "cudaGetErrorName is not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME2(X)\
  printf("%s\n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_NAME3(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          \n", "cudaGetErrorName is not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_NAME3(X)\
  printf("%s\
         \n", hipGetErrorName(X))

// CHECK: /*
// CHECK-NEXT: DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
// CHECK-NEXT: */
// CHECK-NEXT: #define PRINT_ERROR_STR_NAME(X)\
// CHECK-NEXT:   printf("%s\
// CHECK-NEXT:          %s\
// CHECK-NEXT:          \n", "cudaGetErrorString is not supported"/*cudaGetErrorString(X)*/,\
// CHECK-NEXT:          "cudaGetErrorName is not supported"/*cudaGetErrorName(X)*/)
#define PRINT_ERROR_STR_NAME(X)\
  printf("%s\
         %s\
         \n", hipGetErrorString(X),\
         hipGetErrorName(X))

const char *test_function() {
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME(hipGetLastError());
  PRINT_ERROR_STR(hipSuccess);
  PRINT_ERROR_NAME(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR2(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME2(hipGetLastError());
  PRINT_ERROR_STR2(hipSuccess);
  PRINT_ERROR_NAME2(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR3(hipGetLastError());
  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_NAME3(hipGetLastError());
  PRINT_ERROR_STR3(hipSuccess);
  PRINT_ERROR_NAME3(hipSuccess);

  // CHECK: /*
  // CHECK-NEXT: DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
  // CHECK-NEXT: */
  PRINT_ERROR_STR_NAME(hipGetLastError());
  PRINT_ERROR_STR_NAME(hipSuccess);

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:/*
//CHECK-NEXT:DPCT1010:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The call was replaced with 0. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "cudaGetErrorString is not supported"/*cudaGetErrorString(0)*/);
  printf("%s\n", hipGetErrorString(hipGetLastError()));


//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  printf("%s\n", "cudaGetErrorString is not supported"/*cudaGetErrorString(cudaSuccess)*/);
  printf("%s\n", hipGetErrorString(hipSuccess));

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:printf("%s\n", "cudaGetErrorName is not supported"/*cudaGetErrorName(cudaSuccess)*/);
  printf("%s\n", hipGetErrorName(hipSuccess));

//CHECK:/*
//CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL uses exceptions to report errors and does not use the error codes. The original code was commented out and a warning string was inserted. You need to rewrite this code.
//CHECK-NEXT:*/
//CHECK-NEXT:  return "cudaGetErrorName is not supported"/*cudaGetErrorName(cudaSuccess)*/;
  return hipGetErrorName(hipSuccess);
}

