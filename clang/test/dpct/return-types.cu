// RUN: dpct --format-range=none --usm-level=none -out-root %T/return-types %s --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only -fno-delayed-template-parsing
// RUN: FileCheck --input-file %T/return-types/return-types.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/return-types/return-types.dp.cpp -o %T/return-types/return-types.dp.o %}

// CHECK: #include <sycl/sycl.hpp>
// CHECK-NEXT: #include <dpct/dpct.hpp>
// CHECK-NEXT: #include <stdio.h>
// CHECK-EMPTY:

#include <hip/hip_runtime.h>
#include <stdio.h>

// CHECK: #define DEF_BAR dpct::queue_ptr bar() { \
// CHECK-NEXT:   return &dpct::get_out_of_order_queue(); \
// CHECK-NEXT: }
#define DEF_BAR hipStream_t bar() { \
  return 0; \
}
// CHECK: #define DEF_BAR2 dpct::event_ptr bar2() { \
// CHECK-NEXT:   return 0; \
// CHECK-NEXT: }
#define DEF_BAR2 hipEvent_t bar2() { \
  return 0; \
}

DEF_BAR
DEF_BAR2

// need -fno-delayed-template-parsing on windows for
// this migration to pass
// CHECK: template <typename T>
// CHECK-NEXT: dpct::queue_ptr bar() {
// CHECK-NEXT:   return &dpct::get_out_of_order_queue();
// CHECK-NEXT: }
template <typename T>
hipStream_t bar() {
  return 0;
}

// CHECK: template <typename T>
// CHECK-NEXT: dpct::event_ptr bar2() {
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
template <typename T>
hipEvent_t bar2() {
  return 0;
}

// CHECK: dpct::queue_ptr foo() {
hipStream_t foo() {
  // CHECK: return &dpct::get_out_of_order_queue();
  return 0;
}

// CHECK: dpct::event_ptr foo2() {
hipEvent_t foo2() {
  return 0;
}

class S {
  // CHECK: dpct::queue_ptr foo() {
  hipStream_t foo() {
    // CHECK: return &dpct::get_out_of_order_queue();
    return 0;
  }

  // CHECK: dpct::event_ptr foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

class C {
  // CHECK: dpct::queue_ptr foo() {
  hipStream_t foo() {
    // CHECK: return &dpct::get_out_of_order_queue();
    return 0;
  }

  // CHECK: dpct::event_ptr foo2() {
  hipEvent_t foo2() {
    return 0;
  }
};

// CHECK: dpct::queue_ptr *foo(int i) {
hipStream_t *foo(int i) {
  return 0;
}

// CHECK: const dpct::queue_ptr *foo(unsigned i) {
const hipStream_t *foo(unsigned i) {
  return 0;
}

// CHECK: dpct::queue_ptr **foo(char i) {
hipStream_t **foo(char i) {
  return 0;
}

// CHECK: dpct::queue_ptr &foo(short i) {
hipStream_t &foo(short i) {
  hipStream_t s;
  return s;
}

// CHECK: const dpct::queue_ptr &foo(long i) {
const hipStream_t &foo(long i) {
  hipStream_t s;
  return s;
}

// CHECK: dpct::event_ptr *bar(int i) {
hipEvent_t *bar(int i) {
  return 0;
}

// CHECK: const dpct::event_ptr *bar(unsigned i) {
const hipEvent_t *bar(unsigned i) {
  return 0;
}

// CHECK: dpct::event_ptr **bar(char i) {
hipEvent_t **bar(char i) {
  return 0;
}

// CHECK: dpct::event_ptr &bar(short i) {
hipEvent_t &bar(short i) {
  hipEvent_t e;
  return e;
}

// CHECK: const dpct::event_ptr &bar(long i) {
const hipEvent_t &bar(long i) {
  hipEvent_t e;
  return e;
}

