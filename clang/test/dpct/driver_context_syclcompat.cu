// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1, cuda-12.2, cuda-12.3, cuda-12.4
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1, v12.2, v12.3, v12.4
// RUN: dpct --format-range=none -out-root %T/driver_context_syclcompat %s --use-syclcompat --cuda-include-path="%cuda-path/include" -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck %s --match-full-lines --input-file %T/driver_context_syclcompat/driver_context_syclcompat.dp.cpp
// RUN: %if build_lit %{icpx -c -DBUILD_TEST -fsycl %T/driver_context_syclcompat/driver_context_syclcompat.dp.cpp -o %T/driver_context_syclcompat/driver_context_syclcompat.dp.o %}
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main(){
#ifndef BUILD_TEST
  hipDevice_t device;

  // CHECK: int ctx;
  hipCtx_t ctx;

  unsigned int flags = hipDeviceMapHost;
  CUexecAffinityParam* paramsArray;
  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "cuCtxCreate_v3" is not supported with SYCLcompat currently, please adjust the code manually.
  cuCtxCreate_v3(&ctx, paramsArray, 1, flags, device);

  CUctxCreateParams* ctxCreateParams;
  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "cuCtxCreate_v4" is not supported with SYCLcompat currently, please adjust the code manually.
  cuCtxCreate_v4(&ctx, ctxCreateParams, flags, device);  

  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "hipCtxCreate" is not supported with SYCLcompat currently, please adjust the code manually.
  hipCtxCreate(&ctx, hipDeviceLmemResizeToMax, device);  

  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "hipCtxPushCurrent" is not supported with SYCLcompat currently, please adjust the code manually.
  MY_SAFE_CALL(hipCtxPushCurrent(ctx));

  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "hipCtxPopCurrent" is not supported with SYCLcompat currently, please adjust the code manually.
  hipCtxPopCurrent(&ctx);

  // CHECK: ctx = dpct::select_device(device);
  hipDevicePrimaryCtxRetain(&ctx, device);

  // CHECK: MY_SAFE_CALL(DPCT_CHECK_ERROR(dpct::select_device(ctx)));
  MY_SAFE_CALL(hipCtxSetCurrent(ctx));
#endif
  return 0;
}

#ifndef BUILD_TEST
void foo() {
  float *h_A;
  unsigned int numAttributes = 5;

  hipPointer_attribute attributes[] = {
    HIP_POINTER_ATTRIBUTE_MEMORY_TYPE,
    HIP_POINTER_ATTRIBUTE_DEVICE_POINTER,
    HIP_POINTER_ATTRIBUTE_HOST_POINTER,
    HIP_POINTER_ATTRIBUTE_IS_MANAGED,
    HIP_POINTER_ATTRIBUTE_DEVICE_ORDINAL
  };

  hipMemoryType memType;
  void* hostPtr;
  unsigned int isManaged;
  int deviceID;
  hipDeviceptr_t devPtr;

  void* attributeValues[] = {
    &memType,
    &devPtr,
    &hostPtr,
    &isManaged,
    &deviceID
  };

  // CHECK: DPCT1131:{{[0-9]+}}: The migration of "hipDrvPointerGetAttributes" is not supported with SYCLcompat currently, please adjust the code manually.
  hipDrvPointerGetAttributes(
    numAttributes,
    attributes,
    attributeValues,
    (hipDeviceptr_t) h_A
  );
}
#endif
