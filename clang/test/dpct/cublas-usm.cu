// RUN: dpct --format-range=none -out-root %T/cublas-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm/cublas-usm.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublas-usm/cublas-usm.dp.cpp -o %T/cublas-usm/cublas-usm.dp.o %}
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

hipblasHandle_t handle;
int N = 275;
float *h_a, *h_b, *h_c;
const float *d_A_S;
const float *d_B_S;
float *d_C_S;
float alpha_S = 1.0f;
float beta_S = 0.0f;
int trans0 = 0;
int trans1 = 1;
int trans2 = 2;
int fill0 = 0;
int side0 = 0;
int diag0 = 0;
int *result = 0;
const float *x_S = 0;
const float *y_S = 0;

const double *d_A_D;
const double  *d_B_D;
double  *d_C_D;
double alpha_D;
double beta_D;
const double *x_D;
const double *y_D;

const float2 *d_A_C;
const float2  *d_B_C;
float2  *d_C_C;
float2 alpha_C;
float2 beta_C;
const float2 *x_C;
const float2 *y_C;

const double2 *d_A_Z;
const double2  *d_B_Z;
double2  *d_C_Z;
double2 alpha_Z;
double2 beta_Z;
const double2 *x_Z;
const double2 *y_Z;

float* result_S;
double* result_D;
float2* result_C;
double2* result_Z;

int incx, incy, lda, ldb, ldc;

int main() {

  //CHECK:/*
  //CHECK-NEXT:DPCT1018:{{[0-9]+}}: The cublasSetVector was migrated, but due to parameter 11111 equals to parameter 11111 but greater than 1, the generated code performance may be sub-optimal.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int a = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 11111, 11111, 1, 10, sizeof(float)));
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_b, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_c, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:a = DPCT_CHECK_ERROR(dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 100, 100, 100, 100, 10000));
  int a = hipblasSetVector(10, sizeof(float), h_a, 11111, d_C_S, 11111);
  hipblasSetVector(10, sizeof(float), h_b, 1, d_C_S, 1);
  hipblasSetVector(10, sizeof(float), h_c, 1, d_C_S, 1);
  a = hipblasSetMatrix(100, 100, 10000, h_a, 100, d_C_S, 100);


  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed because this functionality is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

  //level 1

  //CHECK:a = [&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(handle->get_queue(), N, x_S, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  a = hipblasIsamax(handle, N, x_S, N, result);
  //CHECK:[&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(handle->get_queue(), N, x_D, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  hipblasIdamax(handle, N, x_D, N, result);
  //CHECK:a = [&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(handle->get_queue(), N, (std::complex<float>*)x_C, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  a = hipblasIcamax(handle, N, x_C, N, result);
  //CHECK:[&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(handle->get_queue(), N, (std::complex<double>*)x_Z, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  hipblasIzamax(handle, N, x_Z, N, result);

  //CHECK:a = [&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(handle->get_queue(), N, x_S, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  a = hipblasIsamin(handle, N, x_S, N, result);
  //CHECK:[&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(handle->get_queue(), N, x_D, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  hipblasIdamin(handle, N, x_D, N, result);
  //CHECK:a = [&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(handle->get_queue(), N, (std::complex<float>*)x_C, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  a = hipblasIcamin(handle, N, x_C, N, result);
  //CHECK:[&]() {
  //CHECK-NEXT:dpct::blas::wrapper_int_to_int64_out res_wrapper_ct4(handle->get_queue(), result);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(handle->get_queue(), N, (std::complex<double>*)x_Z, N, res_wrapper_ct4.get_memory(), oneapi::mkl::index_base::one);
  //CHECK-NEXT:return 0;
  //CHECK-NEXT:}();
  hipblasIzamin(handle, N, x_Z, N, result);

  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotm(handle->get_queue(), N, d_C_S, N, d_C_S, N, const_cast<float*>(x_S)));
  a = hipblasSrotm(handle, N, d_C_S, N, d_C_S, N, x_S);
  //CHECK:oneapi::mkl::blas::column_major::rotm(handle->get_queue(), N, d_C_D, N, d_C_D, N, const_cast<double*>(x_D));
  hipblasDrotm(handle, N, d_C_D, N, d_C_D, N, x_D);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), N, x_S, incx, d_C_S, incy));
  a = hipblasScopy(handle, N, x_S, incx, d_C_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::copy(handle->get_queue(), N, x_D, incx, d_C_D, incy);
  hipblasDcopy(handle, N, x_D, incx, d_C_D, incy);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::copy(handle->get_queue(), N, (std::complex<float>*)x_C, incx, (std::complex<float>*)d_C_C, incy));
  a = hipblasCcopy(handle, N, x_C, incx, d_C_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::copy(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)d_C_Z, incy);
  hipblasZcopy(handle, N, x_Z, incx, d_C_Z, incy);


  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), N, alpha_S, x_S, incx, result_S, incy));
  a = hipblasSaxpy(handle, N, &alpha_S, x_S, incx, result_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::axpy(handle->get_queue(), N, alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, N, &alpha_D, x_D, incx, result_D, incy);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::axpy(handle->get_queue(), N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)result_C, incy));
  a = hipblasCaxpy(handle, N, &alpha_C, x_C, incx, result_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::axpy(handle->get_queue(), N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)result_Z, incy);
  hipblasZaxpy(handle, N, &alpha_Z, x_Z, incx, result_Z, incy);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), N, alpha_S, result_S, incx));
  a = hipblasSscal(handle, N, &alpha_S, result_S, incx);
  // CHECK:oneapi::mkl::blas::column_major::scal(handle->get_queue(), N, alpha_D, result_D, incx);
  hipblasDscal(handle, N, &alpha_D, result_D, incx);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::scal(handle->get_queue(), N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)result_C, incx));
  a = hipblasCscal(handle, N, &alpha_C, result_C, incx);
  // CHECK:oneapi::mkl::blas::column_major::scal(handle->get_queue(), N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)result_Z, incx);
  hipblasZscal(handle, N, &alpha_Z, result_Z, incx);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), N, x_S, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasSnrm2(handle, N, x_S, incx, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), N, x_D, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDnrm2(handle, N, x_D, incx, result_D);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), N, (std::complex<float>*)x_C, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasScnrm2(handle, N, x_C, incx, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDznrm2(handle, N, x_Z, incx, result_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), N, x_S, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasSasum(handle, N, x_S, incx, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), N, x_D, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDasum(handle, N, x_D, incx, result_D);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct4(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), N, (std::complex<float>*)x_C, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasScasum(handle, N, x_C, incx, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct4(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, res_wrapper_ct4.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDzasum(handle, N, x_Z, incx, result_D);

  float *a_S, *b_S, *c_S, *s_S;
  double *a_D, *b_D, *c_D, *s_D;
  float2 *a_C, *b_C, *s_C;
  double2 *a_Z, *b_Z, *s_Z;

  // CHECK: float* a_ct{{[0-9]+}} = a_S;
  // CHECK-NEXT: float* b_ct{{[0-9]+}} = b_S;
  // CHECK-NEXT: float* c_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: float* s_ct{{[0-9]+}} = s_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<float>(4, q_ct1);
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_S;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_S;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_S;
  // CHECK-NEXT: }
  // CHECK-NEXT: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(handle->get_queue(), a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}}));
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_S = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_S = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_S = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  a = hipblasSrotg(handle, a_S, b_S, c_S, s_S);
  // CHECK: double* a_ct{{[0-9]+}} = a_D;
  // CHECK-NEXT: double* b_ct{{[0-9]+}} = b_D;
  // CHECK-NEXT: double* c_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: double* s_ct{{[0-9]+}} = s_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<double>(4, q_ct1);
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_D;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_D;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_D;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_D = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_D = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_D = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  hipblasDrotg(handle, a_D, b_D, c_D, s_D);
  // CHECK: sycl::float2* a_ct{{[0-9]+}} = a_C;
  // CHECK-NEXT: sycl::float2* b_ct{{[0-9]+}} = b_C;
  // CHECK-NEXT: float* c_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: sycl::float2* s_ct{{[0-9]+}} = s_C;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_C, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_C, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(3, q_ct1);
  // CHECK-NEXT:   c_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, q_ct1);
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_C;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_C;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_C;
  // CHECK-NEXT: }
  // CHECK-NEXT: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotg(handle->get_queue(), (std::complex<float>*)a_ct{{[0-9]+}}, (std::complex<float>*)b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, (std::complex<float>*)s_ct{{[0-9]+}}));
  // CHECK-NEXT: if(sycl::get_pointer_type(a_C, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_C, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_C = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_C = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_C = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT:   sycl::free(c_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  a = hipblasCrotg(handle, a_C, b_C, c_S, s_C);
  // CHECK: sycl::double2* a_ct{{[0-9]+}} = a_Z;
  // CHECK-NEXT: sycl::double2* b_ct{{[0-9]+}} = b_Z;
  // CHECK-NEXT: double* c_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: sycl::double2* s_ct{{[0-9]+}} = s_Z;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_Z, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_Z, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(3, q_ct1);
  // CHECK-NEXT:   c_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, q_ct1);
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_Z;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_Z;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_Z;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(handle->get_queue(), (std::complex<double>*)a_ct{{[0-9]+}}, (std::complex<double>*)b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, (std::complex<double>*)s_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_Z, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_Z, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_Z = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_Z = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_Z = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT:   sycl::free(c_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  hipblasZrotg(handle, a_Z, b_Z, c_D, s_Z);

  const float *y1_S;
  const double *y1_D;
  // CHECK: float* d1_ct{{[0-9]+}} = a_S;
  // CHECK-NEXT: float* d2_ct{{[0-9]+}} = b_S;
  // CHECK-NEXT: float* x1_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: float* param_ct{{[0-9]+}} = s_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   d1_ct{{[0-9]+}} = sycl::malloc_shared<float>(8, q_ct1);
  // CHECK-NEXT:   d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *d1_ct{{[0-9]+}} = *a_S;
  // CHECK-NEXT:   *d2_ct{{[0-9]+}} = *b_S;
  // CHECK-NEXT:   *x1_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT: }
  // CHECK-NEXT: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1_S, handle->get_queue()), param_ct{{[0-9]+}}));
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_S = *d1_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_S = *d2_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *x1_ct{{[0-9]+}};
  // CHECK-NEXT:   q_ct1.memcpy(s_S, param_ct{{[0-9]+}}, sizeof(float)*5).wait();
  // CHECK-NEXT:   sycl::free(d1_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  a = hipblasSrotmg(handle, a_S, b_S, c_S, y1_S, s_S);
  // CHECK: double* d1_ct{{[0-9]+}} = a_D;
  // CHECK-NEXT: double* d2_ct{{[0-9]+}} = b_D;
  // CHECK-NEXT: double* x1_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: double* param_ct{{[0-9]+}} = s_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   d1_ct{{[0-9]+}} = sycl::malloc_shared<double>(8, q_ct1);
  // CHECK-NEXT:   d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *d1_ct{{[0-9]+}} = *a_D;
  // CHECK-NEXT:   *d2_ct{{[0-9]+}} = *b_D;
  // CHECK-NEXT:   *x1_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotmg(handle->get_queue(), d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1_D, handle->get_queue()), param_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_queue().get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->get_queue().wait();
  // CHECK-NEXT:   *a_D = *d1_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_D = *d2_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *x1_ct{{[0-9]+}};
  // CHECK-NEXT:   q_ct1.memcpy(s_D, param_ct{{[0-9]+}}, sizeof(double)*5).wait();
  // CHECK-NEXT:   sycl::free(d1_ct{{[0-9]+}}, q_ct1);
  // CHECK-NEXT: }
  hipblasDrotmg(handle, a_D, b_D, c_D, y1_D, s_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct6(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), N, x_S, incx, y_S, incy, res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasSdot(handle, N, x_S, incx, y_S, incy, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct6(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), N, x_D, incx, y_D, incy, res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDdot(handle, N, x_D, incx, y_D, incy, result_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasCdotc(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_Z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasZdotc(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasCdotu(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_Z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasZdotu(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float_out res_wrapper_ct6(handle->get_queue(), result_S);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), N, x_S, incx, y_S, incy, res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasSdot(handle, N, x_S, incx, y_S, incy, result_S);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double_out res_wrapper_ct6(handle->get_queue(), result_D);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dot(handle->get_queue(), N, x_D, incx, y_D, incy, res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasDdot(handle, N, x_D, incx, y_D, incy, result_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasCdotc(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_Z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotc(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasZdotc(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_float2_out res_wrapper_ct6(handle->get_queue(), result_C);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  a = hipblasCdotu(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK: /*
  // CHECK-NEXT: DPCT1034:{{[0-9]+}}: Migrated API does not return an error code. 0 is returned in the lambda. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: [&]() {
  // CHECK-NEXT: dpct::blas::wrapper_double2_out res_wrapper_ct6(handle->get_queue(), result_Z);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::dotu(handle->get_queue(), N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_wrapper_ct6.get_memory());
  // CHECK-NEXT: return 0;
  // CHECK-NEXT: }();
  hipblasZdotu(handle, N, x_Z, incx, y_Z, incy, result_Z);

  //level 2

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, N, N, alpha_S, x_S, lda, y_S, incx, beta_S, result_S, incy));
  a = hipblasSgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::gemv(handle->get_queue(), oneapi::mkl::transpose::nontrans, N, N, alpha_D, x_D, lda, y_D, incx, beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, N, N, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemv(handle->get_queue(), trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, lda, (std::complex<float>*)y_C, incx, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)result_C, incy));
  a = hipblasCgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::gemv(handle->get_queue(), oneapi::mkl::transpose::nontrans, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, lda, (std::complex<double>*)y_Z, incx, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)result_Z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, N, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::ger(handle->get_queue(), N, N, alpha_S, x_S, incx, y_S, incy, result_S, lda));
  a = hipblasSger(handle, N, N, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  // CHECK:oneapi::mkl::blas::column_major::ger(handle->get_queue(), N, N, alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, N, N, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(handle->get_queue(), N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda));
  a = hipblasCgeru(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK:oneapi::mkl::blas::column_major::gerc(handle->get_queue(), N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda);
  hipblasCgerc(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::geru(handle->get_queue(), N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda));
  a = hipblasZgeru(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);
  // CHECK:oneapi::mkl::blas::column_major::gerc(handle->get_queue(), N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda);
  hipblasZgerc(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, lda, (std::complex<float>*)y_C, incx, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)result_C, incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, lda, (std::complex<float>*)y_C, incx, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)result_C, incy);
  a = hipblasCsymv(handle, (hipblasFillMode_t)fill0, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);
  hipblasCsymv(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::symv(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, lda, (std::complex<double>*)y_Z, incx, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)result_Z, incy));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::symv(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, lda, (std::complex<double>*)y_Z, incx, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)result_Z, incy);
  a = hipblasZsymv(handle, (hipblasFillMode_t)fill0, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);
  hipblasZsymv(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)result_C, lda));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)result_C, lda);
  a = hipblasCsyr(handle, (hipblasFillMode_t)fill0, N, &alpha_C, x_C, incx, result_C, lda);
  hipblasCsyr(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_C, x_C, incx, result_C, lda);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)result_Z, lda));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)result_Z, lda);
  a = hipblasZsyr(handle, (hipblasFillMode_t)fill0, N, &alpha_Z, x_Z, incx, result_Z, lda);
  hipblasZsyr(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_Z, x_Z, incx, result_Z, lda);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda);
  a = hipblasCsyr2(handle, (hipblasFillMode_t)fill0, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  hipblasCsyr2(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);

  // CHECK: a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::syr2(handle->get_queue(), fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda));
  // CHECK-NEXT: oneapi::mkl::blas::column_major::syr2(handle->get_queue(), oneapi::mkl::uplo::upper, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda);
  a = hipblasZsyr2(handle, (hipblasFillMode_t)fill0, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);
  hipblasZsyr2(handle, HIPBLAS_FILL_MODE_UPPER, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);

  //level 3

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;

  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(&alpha_S, handle->get_queue()), d_A_S, N, d_B_S, N, dpct::get_value(&beta_S, handle->get_queue()), d_C_S, N));
  a = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(&alpha_D, handle->get_queue()), d_A_D, N, d_B_D, N, dpct::get_value(&beta_D, handle->get_queue()), d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_C, handle->get_queue()), (std::complex<float>*)d_A_C, N, (std::complex<float>*)d_B_C, N, dpct::get_value(&beta_C, handle->get_queue()), (std::complex<float>*)d_C_C, N));
  a = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  //CHECK:oneapi::mkl::blas::column_major::gemm(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, dpct::get_value(&alpha_Z, handle->get_queue()), (std::complex<double>*)d_A_Z, N, (std::complex<double>*)d_B_Z, N, dpct::get_value(&beta_Z, handle->get_queue()), (std::complex<double>*)d_C_Z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);

  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, 16, d_B_S, N, 16, beta_S, d_C_S, N, 16, 10));
  a = hipblasSgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, 16, d_B_S, N, 16, &beta_S, d_C_S, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_D, d_A_D, N, 16, d_B_D, N, 16, beta_D, d_C_D, N, 16, 10);
  hipblasDgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, 16, d_B_D, N, 16, &beta_D, d_C_D, N, 16, 10);
  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)d_A_C, N, 16, (std::complex<float>*)d_B_C, N, 16, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)d_C_C, N, 16, 10));
  a = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, 16, d_B_C, N, 16, &beta_C, d_C_C, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)d_A_Z, N, 16, (std::complex<double>*)d_B_Z, N, 16, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)d_C_Z, N, 16, 10);
  hipblasZgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, 16, d_B_Z, N, 16, &beta_Z, d_C_Z, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(handle->get_queue(), oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H, N, 16, d_B_H, N, 16, beta_H, d_C_H, N, 16, 10);
  hipblasHgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, 16, d_B_H, N, 16, &beta_H, d_C_H, N, 16, 10);

  hipblasOperation_t trans3 = HIPBLAS_OP_N;
  //CHECK:a = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), trans3, trans3, N, N, N, alpha_H, d_A_H, N, d_B_H, N, beta_H, d_C_H, N));
  a = hipblasHgemm(handle, trans3, trans3, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  // CHECK: void *alpha, *beta, *A, *B, *C;
  // CHECK-NEXT: int algo = 0;
  void *alpha, *beta, *A, *B, *C;
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_ALGO0;
  // CHECK: dpct::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, algo);

  float2 alpha_C, beta_C;
  //CHECK: dpct::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_half, N, dpct::library_data_t::real_float);
  //CHECK-NEXT: dpct::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_half, N, B, dpct::library_data_t::real_half, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  //CHECK-NEXT: dpct::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_S, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, &beta_S, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  //CHECK-NEXT: dpct::gemm(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, &alpha_C, A, dpct::library_data_t::complex_float, N, B, dpct::library_data_t::complex_float, N, &beta_C, C, dpct::library_data_t::complex_float, N, dpct::library_data_t::complex_float);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  const float** d_A_S_array;
  const float** d_B_S_array;
  float** d_C_S_array;
  const double** d_A_D_array;
  const double** d_B_D_array;
  double** d_C_D_array;
  const hipComplex** d_A_C_array = 0;
  const hipComplex** d_B_C_array = 0;
  hipComplex** d_C_C_array = 0;
  const hipDoubleComplex** d_A_Z_array = 0;
  const hipDoubleComplex** d_B_Z_array = 0;
  hipDoubleComplex** d_C_Z_array = 0;
  const __half** d_A_H_array = 0;
  const __half** d_B_H_array = 0;
  __half** d_C_H_array = 0;

  // CHECK: a = DPCT_CHECK_ERROR(dpct::gemm_batch(handle->get_queue(), trans3, trans3, N, N, N, &alpha_S, (const void**)d_A_S_array, dpct::library_data_t::real_float, N, (const void**)d_B_S_array, dpct::library_data_t::real_float, N, &beta_S, (void**)d_C_S_array, dpct::library_data_t::real_float, N, 10, dpct::library_data_t::real_float));
  // CHECK-NEXT: dpct::gemm_batch(handle->get_queue(), trans3, trans3, N, N, N, &alpha_D, (const void**)d_A_D_array, dpct::library_data_t::real_double, N, (const void**)d_B_D_array, dpct::library_data_t::real_double, N, &beta_D, (void**)d_C_D_array, dpct::library_data_t::real_double, N, 10, dpct::library_data_t::real_double);
  // CHECK-NEXT: dpct::gemm_batch(handle->get_queue(), trans3, trans3, N, N, N, &alpha_C, (const void**)d_A_C_array, dpct::library_data_t::complex_float, N, (const void**)d_B_C_array, dpct::library_data_t::complex_float, N, &beta_C, (void**)d_C_C_array, dpct::library_data_t::complex_float, N, 10, dpct::library_data_t::complex_float);
  // CHECK-NEXT: dpct::gemm_batch(handle->get_queue(), trans3, trans3, N, N, N, &alpha_Z, (const void**)d_A_Z_array, dpct::library_data_t::complex_double, N, (const void**)d_B_Z_array, dpct::library_data_t::complex_double, N, &beta_Z, (void**)d_C_Z_array, dpct::library_data_t::complex_double, N, 10, dpct::library_data_t::complex_double);
  a = hipblasSgemmBatched(handle, trans3, trans3, N, N, N, &alpha_S, d_A_S_array, N, d_B_S_array, N, &beta_S, d_C_S_array, N, 10);
  hipblasDgemmBatched(handle, trans3, trans3, N, N, N, &alpha_D, d_A_D_array, N, d_B_D_array, N, &beta_D, d_C_D_array, N, 10);
  hipblasCgemmBatched(handle, trans3, trans3, N, N, N, &alpha_C, d_A_C_array, N, d_B_C_array, N, &beta_C, d_C_C_array, N, 10);
  hipblasZgemmBatched(handle, trans3, trans3, N, N, N, &alpha_Z, d_A_Z_array, N, d_B_Z_array, N, &beta_Z, d_C_Z_array, N, 10);

  // CHECK: a = DPCT_CHECK_ERROR(dpct::trsm_batch(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, trans3, oneapi::mkl::diag::unit, N, N, &alpha_S, (const void**)d_A_S_array, dpct::library_data_t::real_float, N, (void**)d_C_S_array, dpct::library_data_t::real_float, N, 10, dpct::library_data_t::real_float));
  // CHECK-NEXT: dpct::trsm_batch(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, trans3, oneapi::mkl::diag::unit, N, N, &alpha_D, (const void**)d_A_D_array, dpct::library_data_t::real_double, N, (void**)d_C_D_array, dpct::library_data_t::real_double, N, 10, dpct::library_data_t::real_double);
  // CHECK-NEXT: dpct::trsm_batch(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, trans3, oneapi::mkl::diag::unit, N, N, &alpha_C, (const void**)d_A_C_array, dpct::library_data_t::complex_float, N, (void**)d_C_C_array, dpct::library_data_t::complex_float, N, 10, dpct::library_data_t::complex_float);
  // CHECK-NEXT: dpct::trsm_batch(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, trans3, oneapi::mkl::diag::unit, N, N, &alpha_Z, (const void**)d_A_Z_array, dpct::library_data_t::complex_double, N, (void**)d_C_Z_array, dpct::library_data_t::complex_double, N, 10, dpct::library_data_t::complex_double);
  a = hipblasStrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_S, d_A_S_array, N, d_C_S_array, N, 10);
  hipblasDtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_D, d_A_D_array, N, d_C_D_array, N, 10);
  hipblasCtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C_array, N, d_C_C_array, N, 10);
  hipblasZtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z_array, N, d_C_Z_array, N, 10);

  //CHECK:a = DPCT_CHECK_ERROR(dpct::trmm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N));
  //CHECK-NEXT:dpct::trmm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_D, d_A_D, N, d_B_D, N, d_C_D, N);
  //CHECK:a = DPCT_CHECK_ERROR(dpct::trmm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N));
  //CHECK-NEXT:dpct::trmm(handle->get_queue(), oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, d_C_Z, N);
  a = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
  hipblasDtrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_D, d_A_D, N, d_B_D, N, d_C_D, N);
  a = hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, d_C_Z, N);

  //CHECK:a = DPCT_CHECK_ERROR(dpct::syrk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N));
  a = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:dpct::syrk(handle->get_queue(), fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);

  // CHECK: if(int stat = DPCT_CHECK_ERROR(dpct::trmm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N))){}
  if(int stat = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N)){}

  // CHECK: if(int stat = DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(&alpha_S, handle->get_queue()), d_A_S, N, d_B_S, N, dpct::get_value(&beta_S, handle->get_queue()), d_C_S, N))){}
  if(int stat = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){}


}

// CHECK:int foo1() try {
// CHECK:  return DPCT_CHECK_ERROR(dpct::trmm(handle->get_queue(), (oneapi::mkl::side)side0, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans0), (oneapi::mkl::diag)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N));
// CHECK-NEXT:}
int foo1(){
  return hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
}

// CHECK:int foo2() try {
// CHECK:  return DPCT_CHECK_ERROR(oneapi::mkl::blas::column_major::gemm(handle->get_queue(), dpct::get_transpose(trans0), dpct::get_transpose(trans1), N, N, N, dpct::get_value(&alpha_S, handle->get_queue()), d_A_S, N, d_B_S, N, dpct::get_value(&beta_S, handle->get_queue()), d_C_S, N));
// CHECK-NEXT:}
int foo2(){
  return hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}

void foo3() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  //CHECK:dpct::syrk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_f, a_f, 3, b_f, 3, beta_f, c_f, 2);
  //CHECK-NEXT:dpct::syrk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_d, a_d, 3, b_d, 3, beta_d, c_d, 2);
  //CHECK-NEXT:dpct::syrk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_c, c_c, 2);
  //CHECK-NEXT:dpct::syrk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_z, c_z, 2);
  //CHECK-NEXT:dpct::herk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_f, c_c, 2);
  //CHECK-NEXT:dpct::herk(handle->get_queue(), oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_d, c_z, 2);
  hipblasSsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_f, a_f, 3, b_f, 3, beta_f, c_f, 2);
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_d, a_d, 3, b_d, 3, beta_d, c_d, 2);
  hipblasCsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_c, c_c, 2);
  hipblasZsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_z, c_z, 2);
  hipblasCherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_f, c_c, 2);
  hipblasZherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_d, c_z, 2);

  int m, n, lda, incx, ldc;
  //CHECK:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, m, n, a_f, lda, x_f, incx, c_f, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, m, n, a_d, lda, x_d, incx, c_d, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, m, n, (std::complex<float>*)a_f, lda, (std::complex<float>*)x_c, incx, (std::complex<float>*)c_c, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm(handle->get_queue(), oneapi::mkl::side::left, m, n, (std::complex<double>*)a_z, lda, (std::complex<double>*)x_z, incx, (std::complex<double>*)c_z, ldc);
  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, m, n, a_f, lda, x_f, incx, c_f, ldc);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, m, n, a_d, lda, x_d, incx, c_d, ldc);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, m, n, (float2*)a_f, lda, x_c, incx, c_c, ldc);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, m, n, a_z, lda, x_z, incx, c_z, ldc);

  //CHECK:oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_f, handle->get_queue()), a_f, lda, dpct::get_value(beta_f, handle->get_queue()), b_f, ldb, c_f, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_d, handle->get_queue()), a_d, lda, dpct::get_value(beta_d, handle->get_queue()), b_d, ldb, c_d, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_c, handle->get_queue()), (std::complex<float>*)a_c, lda, dpct::get_value(beta_c, handle->get_queue()), (std::complex<float>*)b_c, ldb, (std::complex<float>*)c_c, ldc);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::omatadd(handle->get_queue(), oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::trans, m, n, dpct::get_value(alpha_z, handle->get_queue()), (std::complex<double>*)a_z, lda, dpct::get_value(beta_z, handle->get_queue()), (std::complex<double>*)b_z, ldb, (std::complex<double>*)c_z, ldc);
  hipblasSgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_f, a_f, lda, beta_f, b_f, ldb, c_f, ldc);
  hipblasDgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_d, a_d, lda, beta_d, b_d, ldb, c_d, ldc);
  hipblasCgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_c, a_c, lda, beta_c, b_c, ldb, c_c, ldc);
  hipblasZgeam(handle, HIPBLAS_OP_C, HIPBLAS_OP_T, m, n, alpha_z, a_z, lda, beta_z, b_z, ldb, c_z, ldc);
}

void foo4() {
  void *handle;
  hipblasOperation_t trans, transa, transb;
  int m, n, k, lda, ldb, ldc;
  const float *alpha;
  const float *beta;
  const float *a;
  const float *b;
  float *c;
  // CHECK: oneapi::mkl::blas::column_major::gemm(((dpct::blas::descriptor_ptr)handle)->get_queue(), transa, transb, m, n, k, dpct::get_value(alpha, ((dpct::blas::descriptor_ptr)handle)->get_queue()), a, lda, b, ldb, dpct::get_value(beta, ((dpct::blas::descriptor_ptr)handle)->get_queue()), c, ldc);
  hipblasSgemm((hipblasHandle_t)handle, transa, transb, m, n, k, alpha, a, lda, b, ldb, beta, c, ldc);

  int incx, incy;
  const float *x;
  float *y;
  // CHECK: oneapi::mkl::blas::column_major::gemv(((dpct::blas::descriptor_ptr)handle)->get_queue(), trans, m, n, dpct::get_value(alpha, ((dpct::blas::descriptor_ptr)handle)->get_queue()), a, lda, x, incx, dpct::get_value(beta, ((dpct::blas::descriptor_ptr)handle)->get_queue()), y, incy);
  hipblasSgemv((hipblasHandle_t)handle, trans, m, n, alpha, a, lda, x, incx, beta, y, incy);
}
