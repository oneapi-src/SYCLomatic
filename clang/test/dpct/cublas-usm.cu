// RUN: dpct --format-range=none -out-root %T/cublas-usm %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cublas-usm/cublas-usm.dp.cpp --match-full-lines %s
#include <cstdio>
#include <hipblas.h>
#include <hip/hip_runtime.h>

hipblasHandle_t handle;
int N = 275;
float *h_a, *h_b, *h_c;
const float *d_A_S;
const float *d_B_S;
float *d_C_S;
float alpha_S = 1.0f;
float beta_S = 0.0f;
int trans0 = 0;
int trans1 = 1;
int trans2 = 2;
int fill0 = 0;
int side0 = 0;
int diag0 = 0;
int *result = 0;
const float *x_S = 0;
const float *y_S = 0;

const double *d_A_D;
const double  *d_B_D;
double  *d_C_D;
double alpha_D;
double beta_D;
const double *x_D;
const double *y_D;

const float2 *d_A_C;
const float2  *d_B_C;
float2  *d_C_C;
float2 alpha_C;
float2 beta_C;
const float2 *x_C;
const float2 *y_C;

const double2 *d_A_Z;
const double2  *d_B_Z;
double2  *d_C_Z;
double2 alpha_Z;
double2 beta_Z;
const double2 *x_Z;
const double2 *y_Z;

float* result_S;
double* result_D;
float2* result_C;
double2* result_Z;

int incx, incy, lda, ldb, ldc;

int main() {

  //CHECK:/*
  //CHECK-NEXT:DPCT1018:{{[0-9]+}}: The cublasSetVector was migrated, but due to parameter 11111 equals to parameter 11111 but greater than 1, the generated code performance may be sub-optimal.
  //CHECK-NEXT:*/
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:int a = (dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 11111, 11111, 1, 10, sizeof(float)), 0);
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_b, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:dpct::matrix_mem_copy((void*)d_C_S, (void*)h_c, 1, 1, 1, 10, sizeof(float));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (dpct::matrix_mem_copy((void*)d_C_S, (void*)h_a, 100, 100, 100, 100, 10000), 0);
  int a = hipblasSetVector(10, sizeof(float), h_a, 11111, d_C_S, 11111);
  hipblasSetVector(10, sizeof(float), h_b, 1, d_C_S, 1);
  hipblasSetVector(10, sizeof(float), h_c, 1, d_C_S, 1);
  a = hipblasSetMatrix(100, 100, 10000, h_a, 100, d_C_S, 100);


  //CHECK: int mode = 1;
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasGetPointerMode was removed because the function call is redundant in SYCL.
  //CHECK-NEXT: */
  //CHECK-NEXT: /*
  //CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasSetPointerMode was removed because the function call is redundant in SYCL.
  //CHECK-NEXT: */
  hipblasPointerMode_t mode = HIPBLAS_POINTER_MODE_DEVICE;
  hipblasGetPointerMode(handle, &mode);
  hipblasSetPointerMode(handle, HIPBLAS_POINTER_MODE_HOST);

  //level 1

  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::iamax(*handle, N, x_S, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIsamax(handle, N, x_S, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(*handle, N, x_D, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIdamax(handle, N, x_D, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::iamax(*handle, N, (std::complex<float>*)x_C, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIcamax(handle, N, x_C, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamax(*handle, N, (std::complex<double>*)x_Z, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIzamax(handle, N, x_Z, N, result);

  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::iamin(*handle, N, x_S, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIsamin(handle, N, x_S, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(*handle, N, x_D, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIdamin(handle, N, x_D, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::iamin(*handle, N, (std::complex<float>*)x_C, N, res_temp_ptr_ct{{[0-9]+}}).wait(), 0);
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  a = hipblasIcamin(handle, N, x_C, N, result);
  //CHECK:int64_t* res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<int64_t>(1, dpct::get_default_queue());
  //CHECK-NEXT:oneapi::mkl::blas::column_major::iamin(*handle, N, (std::complex<double>*)x_Z, N, res_temp_ptr_ct{{[0-9]+}}).wait();
  //CHECK-NEXT:int res_temp_host_ct{{[0-9]+}} = (int)*res_temp_ptr_ct{{[0-9]+}};
  //CHECK-NEXT:dpct::dpct_memcpy(result, &res_temp_host_ct{{[0-9]+}}, sizeof(int));
  //CHECK-NEXT:sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  hipblasIzamin(handle, N, x_Z, N, result);

  //CHECK:a = (oneapi::mkl::blas::column_major::rotm(*handle, N, d_C_S, N, d_C_S, N, const_cast<float*>(x_S)), 0);
  a = hipblasSrotm(handle, N, d_C_S, N, d_C_S, N, x_S);
  //CHECK:oneapi::mkl::blas::column_major::rotm(*handle, N, d_C_D, N, d_C_D, N, const_cast<double*>(x_D));
  hipblasDrotm(handle, N, d_C_D, N, d_C_D, N, x_D);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::copy(*handle, N, x_S, incx, d_C_S, incy), 0);
  a = hipblasScopy(handle, N, x_S, incx, d_C_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::copy(*handle, N, x_D, incx, d_C_D, incy);
  hipblasDcopy(handle, N, x_D, incx, d_C_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::copy(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)d_C_C, incy), 0);
  a = hipblasCcopy(handle, N, x_C, incx, d_C_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::copy(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)d_C_Z, incy);
  hipblasZcopy(handle, N, x_Z, incx, d_C_Z, incy);


  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::axpy(*handle, N, alpha_S, x_S, incx, result_S, incy), 0);
  a = hipblasSaxpy(handle, N, &alpha_S, x_S, incx, result_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::axpy(*handle, N, alpha_D, x_D, incx, result_D, incy);
  hipblasDaxpy(handle, N, &alpha_D, x_D, incx, result_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::axpy(*handle, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)result_C, incy), 0);
  a = hipblasCaxpy(handle, N, &alpha_C, x_C, incx, result_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::axpy(*handle, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)result_Z, incy);
  hipblasZaxpy(handle, N, &alpha_Z, x_Z, incx, result_Z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::scal(*handle, N, alpha_S, result_S, incx), 0);
  a = hipblasSscal(handle, N, &alpha_S, result_S, incx);
  // CHECK:oneapi::mkl::blas::column_major::scal(*handle, N, alpha_D, result_D, incx);
  hipblasDscal(handle, N, &alpha_D, result_D, incx);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::scal(*handle, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)result_C, incx), 0);
  a = hipblasCscal(handle, N, &alpha_C, result_C, incx);
  // CHECK:oneapi::mkl::blas::column_major::scal(*handle, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)result_Z, incx);
  hipblasZscal(handle, N, &alpha_Z, result_Z, incx);

  // CHECK: float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::nrm2(*handle, N, x_S, incx, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasSnrm2(handle, N, x_S, incx, result_S);
  // CHECK: double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::nrm2(*handle, N, x_D, incx, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDnrm2(handle, N, x_D, incx, result_D);
  // CHECK: float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::nrm2(*handle, N, (std::complex<float>*)x_C, incx, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasScnrm2(handle, N, x_C, incx, result_S);
  // CHECK: double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK:oneapi::mkl::blas::column_major::nrm2(*handle, N, (std::complex<double>*)x_Z, incx, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDznrm2(handle, N, x_Z, incx, result_D);

  // CHECK: float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::asum(*handle, N, x_S, incx, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasSasum(handle, N, x_S, incx, result_S);
  // CHECK: double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, N, x_D, incx, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDasum(handle, N, x_D, incx, result_D);
  // CHECK: float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::asum(*handle, N, (std::complex<float>*)x_C, incx, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasScasum(handle, N, x_C, incx, result_S);
  // CHECK: double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::asum(*handle, N, (std::complex<double>*)x_Z, incx, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDzasum(handle, N, x_Z, incx, result_D);

  float *a_S, *b_S, *c_S, *s_S;
  double *a_D, *b_D, *c_D, *s_D;
  float2 *a_C, *b_C, *s_C;
  double2 *a_Z, *b_Z, *s_Z;

  // CHECK: float* a_ct{{[0-9]+}} = a_S;
  // CHECK-NEXT: float* b_ct{{[0-9]+}} = b_S;
  // CHECK-NEXT: float* c_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: float* s_ct{{[0-9]+}} = s_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<float>(4, dpct::get_default_queue());
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_S;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_S;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_S;
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::rotg(*handle, a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_S = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_S = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_S = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasSrotg(handle, a_S, b_S, c_S, s_S);
  // CHECK: double* a_ct{{[0-9]+}} = a_D;
  // CHECK-NEXT: double* b_ct{{[0-9]+}} = b_D;
  // CHECK-NEXT: double* c_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: double* s_ct{{[0-9]+}} = s_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<double>(4, dpct::get_default_queue());
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   c_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_D;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_D;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_D;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, a_ct{{[0-9]+}}, b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, s_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_D = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_D = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_D = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDrotg(handle, a_D, b_D, c_D, s_D);
  // CHECK: sycl::float2* a_ct{{[0-9]+}} = a_C;
  // CHECK-NEXT: sycl::float2* b_ct{{[0-9]+}} = b_C;
  // CHECK-NEXT: float* c_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: sycl::float2* s_ct{{[0-9]+}} = s_C;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(3, dpct::get_default_queue());
  // CHECK-NEXT:   c_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_C;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_C;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_C;
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::rotg(*handle, (std::complex<float>*)a_ct{{[0-9]+}}, (std::complex<float>*)b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, (std::complex<float>*)s_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(a_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_C = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_C = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_C = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:   sycl::free(c_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasCrotg(handle, a_C, b_C, c_S, s_C);
  // CHECK: sycl::double2* a_ct{{[0-9]+}} = a_Z;
  // CHECK-NEXT: sycl::double2* b_ct{{[0-9]+}} = b_Z;
  // CHECK-NEXT: double* c_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: sycl::double2* s_ct{{[0-9]+}} = s_Z;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   a_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(3, dpct::get_default_queue());
  // CHECK-NEXT:   c_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT:   b_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   s_ct{{[0-9]+}} = a_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   *a_ct{{[0-9]+}} = *a_Z;
  // CHECK-NEXT:   *b_ct{{[0-9]+}} = *b_Z;
  // CHECK-NEXT:   *c_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT:   *s_ct{{[0-9]+}} = *s_Z;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotg(*handle, (std::complex<double>*)a_ct{{[0-9]+}}, (std::complex<double>*)b_ct{{[0-9]+}}, c_ct{{[0-9]+}}, (std::complex<double>*)s_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_Z = *a_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_Z = *b_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *c_ct{{[0-9]+}};
  // CHECK-NEXT:   *s_Z = *s_ct{{[0-9]+}};
  // CHECK-NEXT:   sycl::free(a_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:   sycl::free(c_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasZrotg(handle, a_Z, b_Z, c_D, s_Z);

  const float *y1_S;
  const double *y1_D;
  // CHECK: float* d1_ct{{[0-9]+}} = a_S;
  // CHECK-NEXT: float* d2_ct{{[0-9]+}} = b_S;
  // CHECK-NEXT: float* x1_ct{{[0-9]+}} = c_S;
  // CHECK-NEXT: float* param_ct{{[0-9]+}} = s_S;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   d1_ct{{[0-9]+}} = sycl::malloc_shared<float>(8, dpct::get_default_queue());
  // CHECK-NEXT:   d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *d1_ct{{[0-9]+}} = *a_S;
  // CHECK-NEXT:   *d2_ct{{[0-9]+}} = *b_S;
  // CHECK-NEXT:   *x1_ct{{[0-9]+}} = *c_S;
  // CHECK-NEXT: }
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::rotmg(*handle, d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1_S, *handle), param_ct{{[0-9]+}}), 0);
  // CHECK-NEXT: if(sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_S = *d1_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_S = *d2_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_S = *x1_ct{{[0-9]+}};
  // CHECK-NEXT:   dpct::get_default_queue().memcpy(s_S, param_ct{{[0-9]+}}, sizeof(float)*5).wait();
  // CHECK-NEXT:   sycl::free(d1_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  a = hipblasSrotmg(handle, a_S, b_S, c_S, y1_S, s_S);
  // CHECK: double* d1_ct{{[0-9]+}} = a_D;
  // CHECK-NEXT: double* d2_ct{{[0-9]+}} = b_D;
  // CHECK-NEXT: double* x1_ct{{[0-9]+}} = c_D;
  // CHECK-NEXT: double* param_ct{{[0-9]+}} = s_D;
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   d1_ct{{[0-9]+}} = sycl::malloc_shared<double>(8, dpct::get_default_queue());
  // CHECK-NEXT:   d2_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 1;
  // CHECK-NEXT:   x1_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 2;
  // CHECK-NEXT:   param_ct{{[0-9]+}} = d1_ct{{[0-9]+}} + 3;
  // CHECK-NEXT:   *d1_ct{{[0-9]+}} = *a_D;
  // CHECK-NEXT:   *d2_ct{{[0-9]+}} = *b_D;
  // CHECK-NEXT:   *x1_ct{{[0-9]+}} = *c_D;
  // CHECK-NEXT: }
  // CHECK-NEXT: oneapi::mkl::blas::column_major::rotmg(*handle, d1_ct{{[0-9]+}}, d2_ct{{[0-9]+}}, x1_ct{{[0-9]+}}, dpct::get_value(y1_D, *handle), param_ct{{[0-9]+}});
  // CHECK-NEXT: if(sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(a_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:   handle->wait();
  // CHECK-NEXT:   *a_D = *d1_ct{{[0-9]+}};
  // CHECK-NEXT:   *b_D = *d2_ct{{[0-9]+}};
  // CHECK-NEXT:   *c_D = *x1_ct{{[0-9]+}};
  // CHECK-NEXT:   dpct::get_default_queue().memcpy(s_D, param_ct{{[0-9]+}}, sizeof(double)*5).wait();
  // CHECK-NEXT:   sycl::free(d1_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT: }
  hipblasDrotmg(handle, a_D, b_D, c_D, y1_D, s_D);

  // CHECK:float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dot(*handle, N, x_S, incx, y_S, incy, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasSdot(handle, N, x_S, incx, y_S, incy, result_S);
  // CHECK:double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dot(*handle, N, x_D, incx, y_D, incy, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasDdot(handle, N, x_D, incx, y_D, incy, result_D);

  // CHECK:sycl::float2* res_temp_ptr_ct{{[0-9]+}} = result_C;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dotc(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_C = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasCdotc(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK:sycl::double2* res_temp_ptr_ct{{[0-9]+}} = result_Z;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dotc(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_Z = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasZdotc(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK:sycl::float2* res_temp_ptr_ct{{[0-9]+}} = result_C;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dotu(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_C = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasCdotu(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK:sycl::double2* res_temp_ptr_ct{{[0-9]+}} = result_Z;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dotu(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_Z = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasZdotu(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK:float* res_temp_ptr_ct{{[0-9]+}} = result_S;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<float>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dot(*handle, N, x_S, incx, y_S, incy, res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_S, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_S = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasSdot(handle, N, x_S, incx, y_S, incy, result_S);
  // CHECK:double* res_temp_ptr_ct{{[0-9]+}} = result_D;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<double>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dot(*handle, N, x_D, incx, y_D, incy, res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_D, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_D = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasDdot(handle, N, x_D, incx, y_D, incy, result_D);

  // CHECK:sycl::float2* res_temp_ptr_ct{{[0-9]+}} = result_C;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dotc(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_C = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasCdotc(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK:sycl::double2* res_temp_ptr_ct{{[0-9]+}} = result_Z;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dotc(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_Z = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasZdotc(handle, N, x_Z, incx, y_Z, incy, result_Z);

  // CHECK:sycl::float2* res_temp_ptr_ct{{[0-9]+}} = result_C;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::float2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:/*
  // CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT:*/
  // CHECK-NEXT:a = (oneapi::mkl::blas::column_major::dotu(*handle, N, (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)res_temp_ptr_ct{{[0-9]+}}), 0);
  // CHECK-NEXT:if(sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_C, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_C = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  a = hipblasCdotu(handle, N, x_C, incx, y_C, incy, result_C);
  // CHECK:sycl::double2* res_temp_ptr_ct{{[0-9]+}} = result_Z;
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  res_temp_ptr_ct{{[0-9]+}} = sycl::malloc_shared<sycl::double2>(1, dpct::get_default_queue());
  // CHECK-NEXT:}
  // CHECK-NEXT:oneapi::mkl::blas::column_major::dotu(*handle, N, (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)res_temp_ptr_ct{{[0-9]+}});
  // CHECK-NEXT:if(sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::device && sycl::get_pointer_type(result_Z, handle->get_context())!=sycl::usm::alloc::shared) {
  // CHECK-NEXT:  handle->wait();
  // CHECK-NEXT:  *result_Z = *res_temp_ptr_ct{{[0-9]+}};
  // CHECK-NEXT:  sycl::free(res_temp_ptr_ct{{[0-9]+}}, dpct::get_default_queue());
  // CHECK-NEXT:}
  hipblasZdotu(handle, N, x_Z, incx, y_Z, incy, result_Z);

  //level 2

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::gemv(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, N, N, alpha_S, x_S, lda, y_S, incx, beta_S, result_S, incy), 0);
  a = hipblasSgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_S, x_S, lda, y_S, incx, &beta_S, result_S, incy);
  // CHECK:oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, N, N, alpha_D, x_D, lda, y_D, incx, beta_D, result_D, incy);
  hipblasDgemv(handle, HIPBLAS_OP_N, N, N, &alpha_D, x_D, lda, y_D, incx, &beta_D, result_D, incy);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::gemv(*handle, trans2==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans2, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, lda, (std::complex<float>*)y_C, incx, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)result_C, incy), 0);
  a = hipblasCgemv(handle, (hipblasOperation_t)trans2, N, N, &alpha_C, x_C, lda, y_C, incx, &beta_C, result_C, incy);
  // CHECK:oneapi::mkl::blas::column_major::gemv(*handle, oneapi::mkl::transpose::nontrans, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, lda, (std::complex<double>*)y_Z, incx, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)result_Z, incy);
  hipblasZgemv(handle, HIPBLAS_OP_N, N, N, &alpha_Z, x_Z, lda, y_Z, incx, &beta_Z, result_Z, incy);

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::ger(*handle, N, N, alpha_S, x_S, incx, y_S, incy, result_S, lda), 0);
  a = hipblasSger(handle, N, N, &alpha_S, x_S, incx, y_S, incy, result_S, lda);
  // CHECK:oneapi::mkl::blas::column_major::ger(*handle, N, N, alpha_D, x_D, incx, y_D, incy, result_D, lda);
  hipblasDger(handle, N, N, &alpha_D, x_D, incx, y_D, incy, result_D, lda);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::geru(*handle, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda), 0);
  a = hipblasCgeru(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK:oneapi::mkl::blas::column_major::gerc(*handle, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)x_C, incx, (std::complex<float>*)y_C, incy, (std::complex<float>*)result_C, lda);
  hipblasCgerc(handle, N, N, &alpha_C, x_C, incx, y_C, incy, result_C, lda);
  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::geru(*handle, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda), 0);
  a = hipblasZgeru(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);
  // CHECK:oneapi::mkl::blas::column_major::gerc(*handle, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)x_Z, incx, (std::complex<double>*)y_Z, incy, (std::complex<double>*)result_Z, lda);
  hipblasZgerc(handle, N, N, &alpha_Z, x_Z, incx, y_Z, incy, result_Z, lda);








  //level 3

  __half *d_A_H = 0;
  __half *d_B_H = 0;
  __half *d_C_H = 0;
  __half alpha_H;
  __half beta_H;

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N), 0);
  a = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_D, d_A_D, N, d_B_D, N, beta_D, d_C_D, N);
  hipblasDgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)d_A_C, N, (std::complex<float>*)d_B_C, N, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)d_C_C, N), 0);
  a = hipblasCgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, d_B_C, N, &beta_C, d_C_C, N);
  //CHECK:oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)d_A_Z, N, (std::complex<double>*)d_B_Z, N, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)d_C_Z, N);
  hipblasZgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, &beta_Z, d_C_Z, N);

  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, 16, d_B_S, N, 16, beta_S, d_C_S, N, 16, 10), 0);
  a = hipblasSgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, 16, d_B_S, N, 16, &beta_S, d_C_S, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_D, d_A_D, N, 16, d_B_D, N, 16, beta_D, d_C_D, N, 16, 10);
  hipblasDgemmStridedBatched(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_D, d_A_D, N, 16, d_B_D, N, 16, &beta_D, d_C_D, N, 16, 10);
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)d_A_C, N, 16, (std::complex<float>*)d_B_C, N, 16, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)d_C_C, N, 16, 10), 0);
  a = hipblasCgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_C, d_A_C, N, 16, d_B_C, N, 16, &beta_C, d_C_C, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)d_A_Z, N, 16, (std::complex<double>*)d_B_Z, N, 16, std::complex<double>(beta_Z.x(), beta_Z.y()), (std::complex<double>*)d_C_Z, N, 16, 10);
  hipblasZgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_Z, d_A_Z, N, 16, d_B_Z, N, 16, &beta_Z, d_C_Z, N, 16, 10);
  //CHECK:oneapi::mkl::blas::column_major::gemm_batch(*handle, oneapi::mkl::transpose::nontrans, oneapi::mkl::transpose::nontrans, N, N, N, alpha_H, d_A_H, N, 16, d_B_H, N, 16, beta_H, d_C_H, N, 16, 10);
  hipblasHgemmStridedBatched(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, N, N, &alpha_H, d_A_H, N, 16, d_B_H, N, 16, &beta_H, d_C_H, N, 16, 10);

  hipblasOperation_t trans3 = HIPBLAS_OP_N;
  //CHECK:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::gemm(*handle, trans3, trans3, N, N, N, alpha_H, d_A_H, N, d_B_H, N, beta_H, d_C_H, N), 0);
  a = hipblasHgemm(handle, trans3, trans3, N, N, N, &alpha_H, d_A_H, N, d_B_H, N, &beta_H, d_C_H, N);

  // CHECK: void *alpha, *beta, *A, *B, *C;
  // CHECK-NEXT: int algo = 0;
  void *alpha, *beta, *A, *B, *C;
  hipblasGemmAlgo_t algo = CUBLAS_GEMM_ALGO0;
  // CHECK: dpct::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha, A, dpct::library_data_t::real_float, N, B, dpct::library_data_t::real_float, N, beta, C, dpct::library_data_t::real_float, N, dpct::library_data_t::real_float);
  hipblasGemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, alpha, A, HIP_R_32F, N, B, HIP_R_32F, N, beta, C, HIP_R_32F, N, HIP_R_32F, algo);

  float2 alpha_C, beta_C;
  // CHECK: oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, sycl::vec<float, 1>{alpha_S}.convert<sycl::half, sycl::rounding_mode::automatic>()[0], (sycl::half*)A, N, (sycl::half*)B, N, sycl::vec<float, 1>{beta_S}.convert<sycl::half, sycl::rounding_mode::automatic>()[0], (sycl::half*)C, N);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_S, (sycl::half*)A, N, (sycl::half*)B, N, beta_S, (float*)C, N);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, alpha_S, (float*)A, N, (float*)B, N, beta_S, (float*)C, N);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::gemm(*handle, oneapi::mkl::transpose::conjtrans, oneapi::mkl::transpose::conjtrans, N, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)A, N, (std::complex<float>*)B, N, std::complex<float>(beta_C.x(), beta_C.y()), (std::complex<float>*)C, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_16F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_16F, N, B, HIP_R_16F, N, &beta_S, C, HIP_R_32F, N);
  cublasSgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_S, A, HIP_R_32F, N, B, HIP_R_32F, N, &beta_S, C, HIP_R_32F, N);
  cublasCgemmEx(handle, HIPBLAS_OP_C, HIPBLAS_OP_C, N, N, N, &alpha_C, A, HIP_C_32F, N, B, HIP_C_32F, N, &beta_C, C, HIP_C_32F, N);

  const float** d_A_S_array;
  const float** d_B_S_array;
  float** d_C_S_array;
  const double** d_A_D_array;
  const double** d_B_D_array;
  double** d_C_D_array;
  const hipComplex** d_A_C_array = 0;
  const hipComplex** d_B_C_array = 0;
  hipComplex** d_C_C_array = 0;
  const hipDoubleComplex** d_A_Z_array = 0;
  const hipDoubleComplex** d_B_Z_array = 0;
  hipDoubleComplex** d_C_Z_array = 0;
  const __half** d_A_H_array = 0;
  const __half** d_B_H_array = 0;
  __half** d_C_H_array = 0;

  // CHECK: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, k_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, ldc_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: float alpha_ct{{[0-9]+}} = alpha_S, beta_ct{{[0-9]+}} = beta_S;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::gemm_batch(*handle, &trans3, &trans3, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &k_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, d_A_S_array, &lda_ct{{[0-9]+}}, d_B_S_array, &ldb_ct{{[0-9]+}}, &beta_ct{{[0-9]+}}, d_C_S_array, &ldc_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {}), 0);
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, k_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, ldc_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: double alpha_ct{{[0-9]+}} = alpha_D, beta_ct{{[0-9]+}} = beta_D;
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, &trans3, &trans3, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &k_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, d_A_D_array, &lda_ct{{[0-9]+}}, d_B_D_array, &ldb_ct{{[0-9]+}}, &beta_ct{{[0-9]+}}, d_C_D_array, &ldc_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, k_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, ldc_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: std::complex<float> alpha_ct{{[0-9]+}} = std::complex<float>(alpha_C.x(), alpha_C.y()), beta_ct{{[0-9]+}} = std::complex<float>(beta_C.x(), beta_C.y());
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, &trans3, &trans3, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &k_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, (const std::complex<float>**)d_A_C_array, &lda_ct{{[0-9]+}}, (const std::complex<float>**)d_B_C_array, &ldb_ct{{[0-9]+}}, &beta_ct{{[0-9]+}}, (std::complex<float>**)d_C_C_array, &ldc_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, k_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, ldc_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: std::complex<double> alpha_ct{{[0-9]+}} = std::complex<double>(alpha_Z.x(), alpha_Z.y()), beta_ct{{[0-9]+}} = std::complex<double>(beta_Z.x(), beta_Z.y());
  // CHECK-NEXT: oneapi::mkl::blas::column_major::gemm_batch(*handle, &trans3, &trans3, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &k_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, (const std::complex<double>**)d_A_Z_array, &lda_ct{{[0-9]+}}, (const std::complex<double>**)d_B_Z_array, &ldb_ct{{[0-9]+}}, &beta_ct{{[0-9]+}}, (std::complex<double>**)d_C_Z_array, &ldc_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  a = hipblasSgemmBatched(handle, trans3, trans3, N, N, N, &alpha_S, d_A_S_array, N, d_B_S_array, N, &beta_S, d_C_S_array, N, 10);
  hipblasDgemmBatched(handle, trans3, trans3, N, N, N, &alpha_D, d_A_D_array, N, d_B_D_array, N, &beta_D, d_C_D_array, N, 10);
  hipblasCgemmBatched(handle, trans3, trans3, N, N, N, &alpha_C, d_A_C_array, N, d_B_C_array, N, &beta_C, d_C_C_array, N, 10);
  hipblasZgemmBatched(handle, trans3, trans3, N, N, N, &alpha_Z, d_A_Z_array, N, d_B_Z_array, N, &beta_Z, d_C_Z_array, N, 10);

  // CHECK: oneapi::mkl::side side_ct{{[0-9]+}} = oneapi::mkl::side::left;
  // CHECK-NEXT: oneapi::mkl::uplo uplo_ct{{[0-9]+}} = oneapi::mkl::uplo::lower;
  // CHECK-NEXT: oneapi::mkl::diag diag_ct{{[0-9]+}} = oneapi::mkl::diag::unit;
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: float alpha_ct{{[0-9]+}} = alpha_S;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: a = (oneapi::mkl::blas::column_major::trsm_batch(*handle, &side_ct{{[0-9]+}}, &uplo_ct{{[0-9]+}}, &trans3, &diag_ct{{[0-9]+}}, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, d_A_S_array, &lda_ct{{[0-9]+}}, d_C_S_array, &ldb_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {}), 0);
  // CHECK-NEXT: oneapi::mkl::side side_ct{{[0-9]+}} = oneapi::mkl::side::left;
  // CHECK-NEXT: oneapi::mkl::uplo uplo_ct{{[0-9]+}} = oneapi::mkl::uplo::lower;
  // CHECK-NEXT: oneapi::mkl::diag diag_ct{{[0-9]+}} = oneapi::mkl::diag::unit;
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: double alpha_ct{{[0-9]+}} = alpha_D;
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm_batch(*handle, &side_ct{{[0-9]+}}, &uplo_ct{{[0-9]+}}, &trans3, &diag_ct{{[0-9]+}}, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, d_A_D_array, &lda_ct{{[0-9]+}}, d_C_D_array, &ldb_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  // CHECK-NEXT: oneapi::mkl::side side_ct{{[0-9]+}} = oneapi::mkl::side::left;
  // CHECK-NEXT: oneapi::mkl::uplo uplo_ct{{[0-9]+}} = oneapi::mkl::uplo::lower;
  // CHECK-NEXT: oneapi::mkl::diag diag_ct{{[0-9]+}} = oneapi::mkl::diag::unit;
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: std::complex<float> alpha_ct{{[0-9]+}} = std::complex<float>(alpha_C.x(), alpha_C.y());
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm_batch(*handle, &side_ct{{[0-9]+}}, &uplo_ct{{[0-9]+}}, &trans3, &diag_ct{{[0-9]+}}, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, (const std::complex<float>**)d_A_C_array, &lda_ct{{[0-9]+}}, (std::complex<float>**)d_C_C_array, &ldb_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  // CHECK-NEXT: oneapi::mkl::side side_ct{{[0-9]+}} = oneapi::mkl::side::left;
  // CHECK-NEXT: oneapi::mkl::uplo uplo_ct{{[0-9]+}} = oneapi::mkl::uplo::lower;
  // CHECK-NEXT: oneapi::mkl::diag diag_ct{{[0-9]+}} = oneapi::mkl::diag::unit;
  // CHECK-NEXT: int64_t m_ct{{[0-9]+}} = N, n_ct{{[0-9]+}} = N, lda_ct{{[0-9]+}} = N, ldb_ct{{[0-9]+}} = N, group_size_ct{{[0-9]+}} = 10;
  // CHECK-NEXT: std::complex<double> alpha_ct{{[0-9]+}} = std::complex<double>(alpha_Z.x(), alpha_Z.y());
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trsm_batch(*handle, &side_ct{{[0-9]+}}, &uplo_ct{{[0-9]+}}, &trans3, &diag_ct{{[0-9]+}}, &m_ct{{[0-9]+}}, &n_ct{{[0-9]+}}, &alpha_ct{{[0-9]+}}, (const std::complex<double>**)d_A_Z_array, &lda_ct{{[0-9]+}}, (std::complex<double>**)d_C_Z_array, &ldb_ct{{[0-9]+}}, 1, &group_size_ct{{[0-9]+}}, {});
  a = hipblasStrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_S, d_A_S_array, N, d_C_S_array, N, 10);
  hipblasDtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_D, d_A_D_array, N, d_C_D_array, N, 10);
  hipblasCtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C_array, N, d_C_C_array, N, 10);
  hipblasZtrsmBatched(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, trans3, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z_array, N, d_C_Z_array, N, 10);

  //CHECK:dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::trmm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N), 0);
  a = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
  //CHECK:dpct::matrix_mem_copy(d_C_D, d_B_D, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, N, N, alpha_D, d_A_D, N, d_C_D, N);
  hipblasDtrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_D, d_A_D, N, d_B_D, N, d_C_D, N);
  //CHECK:dpct::matrix_mem_copy(d_C_C, d_B_C, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  //CHECK-NEXT:*/
  //CHECK-NEXT:a = (oneapi::mkl::blas::column_major::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, N, N, std::complex<float>(alpha_C.x(), alpha_C.y()), (std::complex<float>*)d_A_C, N, (std::complex<float>*)d_C_C, N), 0);
  a = hipblasCtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_C, d_A_C, N, d_B_C, N, d_C_C, N);
  //CHECK:dpct::matrix_mem_copy(d_C_Z, d_B_Z, N, N, N, N, dpct::device_to_device, *handle);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::trmm(*handle, oneapi::mkl::side::left, oneapi::mkl::uplo::lower, oneapi::mkl::transpose::nontrans, oneapi::mkl::diag::unit, N, N, std::complex<double>(alpha_Z.x(), alpha_Z.y()), (std::complex<double>*)d_A_Z, N, (std::complex<double>*)d_C_Z, N);
  hipblasZtrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_UNIT, N, N, &alpha_Z, d_A_Z, N, d_B_Z, N, d_C_Z, N);


  //CHECK:a = (dpct::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N), 0);
  a = hipblasSsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
  //CHECK:dpct::syrk(*handle, fill0 == 0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, dpct::get_transpose(trans1), N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);
  hipblasDsyrkx(handle, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans1, N, N, &alpha_D, d_A_D, N, d_B_D, N, &beta_D, d_C_D, N);



  // CHECK: dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
  // CHECK-NEXT: oneapi::mkl::blas::column_major::trmm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N);
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in an if statement. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if(int stat = 0){}
  if(int stat = hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N)){}

  // CHECK: /*
  // CHECK-NEXT: DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
  // CHECK-NEXT: */
  // CHECK-NEXT: if(int stat = (oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N), 0)){}
  if(int stat = hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N)){}


}

// CHECK: int foo1() try {
// CHECK-NEXT:   dpct::matrix_mem_copy(d_C_S, d_B_S, N, N, N, N, dpct::device_to_device, *handle);
// CHECK-NEXT:   oneapi::mkl::blas::column_major::trmm(*handle, (oneapi::mkl::side)side0, fill0==0 ? oneapi::mkl::uplo::lower : oneapi::mkl::uplo::upper, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, (oneapi::mkl::diag)diag0, N, N, alpha_S, d_A_S, N, d_C_S, N);
// CHECK-NEXT:   /*
// CHECK-NEXT:   DPCT1041:{{[0-9]+}}: SYCL uses exceptions to report errors, it does not use error codes. 0 is used instead of an error code in a return statement. You may need to rewrite this code.
// CHECK-NEXT:   */
// CHECK-NEXT:   return 0;
// CHECK-NEXT: }
int foo1(){
  return hipblasStrmm(handle, (hipblasSideMode_t)side0, (hipblasFillMode_t)fill0, (hipblasOperation_t)trans0, (hipblasDiagType_t)diag0, N, N, &alpha_S, d_A_S, N, d_B_S, N, d_C_S, N);
}

// CHECK:int foo2() try {
// CHECK-NEXT:  /*
// CHECK-NEXT:  DPCT1003:{{[0-9]+}}: Migrated API does not return error code. (*, 0) is inserted. You may need to rewrite this code.
// CHECK-NEXT:  */
// CHECK-NEXT:  return (oneapi::mkl::blas::column_major::gemm(*handle, trans0==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans0, trans1==2 ? oneapi::mkl::transpose::conjtrans : (oneapi::mkl::transpose)trans1, N, N, N, alpha_S, d_A_S, N, d_B_S, N, beta_S, d_C_S, N), 0);
// CHECK-NEXT:}
int foo2(){
  return hipblasSgemm(handle, (hipblasOperation_t)trans0, (hipblasOperation_t)trans1, N, N, N, &alpha_S, d_A_S, N, d_B_S, N, &beta_S, d_C_S, N);
}

void foo3() {
  hipblasHandle_t handle;
  float   *a_f, *b_f, *x_f, *c_f, *alpha_f, *beta_f;
  double  *a_d, *b_d, *x_d, *c_d, *alpha_d, *beta_d;
  float2  *a_c, *b_c, *x_c, *c_c, *alpha_c, *beta_c;
  double2 *a_z, *b_z, *x_z, *c_z, *alpha_z, *beta_z;

  //CHECK:dpct::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_f, a_f, 3, b_f, 3, beta_f, c_f, 2);
  //CHECK-NEXT:dpct::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_d, a_d, 3, b_d, 3, beta_d, c_d, 2);
  //CHECK-NEXT:dpct::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_c, c_c, 2);
  //CHECK-NEXT:dpct::syrk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::conjtrans, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_z, c_z, 2);
  //CHECK-NEXT:dpct::herk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_f, c_c, 2);
  //CHECK-NEXT:dpct::herk(*handle, oneapi::mkl::uplo::upper, oneapi::mkl::transpose::trans, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_d, c_z, 2);
  hipblasSsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_f, a_f, 3, b_f, 3, beta_f, c_f, 2);
  hipblasDsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_d, a_d, 3, b_d, 3, beta_d, c_d, 2);
  hipblasCsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_c, c_c, 2);
  hipblasZsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_C, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_z, c_z, 2);
  hipblasCherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_c, a_c, 3, b_c, 3, beta_f, c_c, 2);
  hipblasZherkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha_z, a_z, 3, b_z, 3, beta_d, c_z, 2);

  //CHECK:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_f, 2, 0, x_f, 1, 0, c_f, 2, 0, 1);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, a_d, 2, 0, x_d, 1, 0, c_d, 2, 0, 1);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, (std::complex<float>*)a_c, 2, 0, (std::complex<float>*)x_c, 1, 0, (std::complex<float>*)c_c, 2, 0, 1);
  //CHECK-NEXT:oneapi::mkl::blas::column_major::dgmm_batch(*handle, oneapi::mkl::side::left, 2, 2, (std::complex<double>*)a_z, 2, 0, (std::complex<double>*)x_z, 1, 0, (std::complex<double>*)c_z, 2, 0, 1);
  hipblasSdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_f, 2, x_f, 1, c_f, 2);
  hipblasDdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_d, 2, x_d, 1, c_d, 2);
  hipblasCdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_c, 2, x_c, 1, c_c, 2);
  hipblasZdgmm(handle, HIPBLAS_SIDE_LEFT, 2, 2, a_z, 2, x_z, 1, c_z, 2);
}
