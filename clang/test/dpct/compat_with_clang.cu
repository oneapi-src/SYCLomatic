#include "hip/hip_runtime.h"
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none -out-root %T/compat_with_clang %s --cuda-include-path="%cuda-path/include" --stop-on-parse-err
// RUN: FileCheck %s --match-full-lines --input-file %T/compat_with_clang/compat_with_clang.dp.cpp
// RUN: %if build_lit %{icpx -c -fsycl %T/compat_with_clang/compat_with_clang.dp.cpp -o %T/compat_with_clang/compat_with_clang.dp.o %}

#include "hip/hip_fp16.h"
#include <cstdint>

// CHECK: inline void foo1(sycl::half2 *array, sycl::half a) {
// CHECK-NEXT:   array[dpct::reverse_bits<unsigned int>(123)] = {a, sycl::vec<float, 1>(2.3f).convert<sycl::half, sycl::rounding_mode::automatic>()[0]};
// CHECK-NEXT: }
__device__ inline void foo1(__half2 *array, __half a) {
  array[__brev(123)] = {a, __float2half(2.3f)};
}

// CHECK: void foo2(int a, int b) {
// CHECK-NEXT:   dpct::dim3 block{dpct::min(512, uint32_t(a * b))};
// CHECK-NEXT: }
void foo2(int a, int b) {
  dim3 block{min(512, uint32_t(a * b))};
}

template <class T1, class T2> struct AAAAA {
  template <class T3> void foo(T3 x);
};

// CHECK: template <typename T4, typename T5>
// CHECK-NEXT: template <typename T6>
// CHECK-NEXT: void AAAAA<T4, T5>::foo(T6 x) {}
template <typename T4, typename T5>
template <typename T6>
void AAAAA<T4, T5>::foo<T6>(T6 x) {}

class test1 {
  public:
    __host__ __device__ test1();
  private:
    int state;
};

inline test1::test1() : state(0) {
  printf("hello1\n");
}

__host__ __device__ int test2();

int test2() {
  printf("hello2\n");
  return 0;
}

__global__ void kernel() {
  test1 t1;
  test2();
}

void foo3() {
  kernel<<<1, 1>>>();
  hipDeviceSynchronize();
}
