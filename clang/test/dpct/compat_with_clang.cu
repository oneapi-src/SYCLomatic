#include "hip/hip_runtime.h"
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none -out-root %T/compat_with_clang %s --cuda-include-path="%cuda-path/include" --stop-on-parse-err
// RUN: FileCheck %s --match-full-lines --input-file %T/compat_with_clang/compat_with_clang.dp.cpp

#include "hip/hip_fp16.h"

// CHECK: inline void foo1(sycl::half2 *array, sycl::half a) {
// CHECK-NEXT:   array[dpct::reverse_bits<unsigned int>(123)] = {a, sycl::vec<float, 1>{2.3f}.convert<sycl::half, sycl::rounding_mode::automatic>()[0]};
// CHECK-NEXT: }
__device__ inline void foo1(__half2 *array, __half a) {
  array[__brev(123)] = {a, __float2half(2.3f)};
}


template <class T1, class T2> struct AAAAA {
  template <class T3> void foo(T3 x);
};

// CHECK: template <typename T4, typename T5>
// CHECK-NEXT: template <typename T6>
// CHECK-NEXT: void AAAAA<T4, T5>::foo(T6 x) {}
template <typename T4, typename T5>
template <typename T6>
void AAAAA<T4, T5>::foo<T6>(T6 x) {}
