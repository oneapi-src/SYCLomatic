#include "hip/hip_runtime.h"
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none -out-root %T/compat_with_clang %s --cuda-include-path="%cuda-path/include" --stop-on-parse-err
// RUN: FileCheck %s --match-full-lines --input-file %T/compat_with_clang/compat_with_clang.dp.cpp

#include "hip/hip_fp16.h"

// CHECK: inline void foo1(sycl::half2 *array, sycl::half a) {
// CHECK-NEXT:   array[10] = {a, a};
// CHECK-NEXT: }
__device__ inline void foo1(__half2 *array, __half a) {
  array[10] = {a, a};
}
