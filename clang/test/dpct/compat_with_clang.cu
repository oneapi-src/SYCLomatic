#include "hip/hip_runtime.h"
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1
// RUN: dpct --format-range=none -out-root %T/compat_with_clang %s --cuda-include-path="%cuda-path/include" --stop-on-parse-err
// RUN: FileCheck %s --match-full-lines --input-file %T/compat_with_clang/compat_with_clang.dp.cpp

#include "hip/hip_fp16.h"

// CHECK: inline void foo1(sycl::half2 *array, sycl::half a) {
// CHECK-NEXT:   array[dpct::reverse_bits<unsigned int>(123)] = {a, sycl::vec<float, 1>{2.3f}.convert<sycl::half, sycl::rounding_mode::automatic>()[0]};
// CHECK-NEXT: }
__device__ inline void foo1(__half2 *array, __half a) {
  array[__brev(123)] = {a, __float2half(2.3f)};
}
