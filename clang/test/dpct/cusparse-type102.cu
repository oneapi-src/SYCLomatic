// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none --out-root %T/cusparse-type102 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/cusparse-type102/cusparse-type102.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cusparse-type102/cusparse-type102.dp.cpp -o %T/cusparse-type102/cusparse-type102.dp.o %}
#include <cstdio>
#include <hip/hip_runtime.h>
#include <hipsparse.h>

// CUSPARSE_STATUS_NOT_SUPPORTED is available since v10.2.
int main() {
  //CHECK: int a6;
  //CHECK-NEXT: a6 = 10;
  hipsparseStatus_t a6;
  a6 = HIPSPARSE_STATUS_NOT_SUPPORTED;

  //CHECK:/*
  //CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
  //CHECK-NEXT:*/
  //CHECK-NEXT:printf("Error string: %s", dpct::get_error_string_dummy(a6));
  //CHECK-NEXT:/*
  //CHECK-NEXT:DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
  //CHECK-NEXT:*/
  //CHECK-NEXT:printf("Error name: %s", dpct::get_error_string_dummy(a6));
  printf("Error string: %s", hipsparseGetErrorString(a6));
  printf("Error name: %s", hipsparseGetErrorName(a6));

  //CHECK:dpct::library_data_t b1 = dpct::library_data_t::real_uint16;
  //CHECK-NEXT:b1 = dpct::library_data_t::real_int32;
  //CHECK-NEXT:b1 = dpct::library_data_t::real_int64;
  //CHECK-NEXT:oneapi::mkl::layout b2 = oneapi::mkl::layout::col_major;
  //CHECK-NEXT:b2 = oneapi::mkl::layout::row_major;
  hipsparseIndexType_t b1 = HIPSPARSE_INDEX_16U;
  b1 = HIPSPARSE_INDEX_32I;
  b1 = HIPSPARSE_INDEX_64I;
  hipsparseOrder_t b2 = HIPSPARSE_ORDER_COL;
  b2 = HIPSPARSE_ORDER_ROW;

  return 0;
}

//CHECK:void foo(int err) {
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::get_error_string_dummy(err);
//CHECK-NEXT:  /*
//CHECK-NEXT:  DPCT1009:{{[0-9]+}}: SYCL reports errors using exceptions and does not use error codes. Please replace the "get_error_string_dummy(...)" with a real error-handling function.
//CHECK-NEXT:  */
//CHECK-NEXT:  dpct::get_error_string_dummy({{[0-9]+}});
//CHECK-NEXT:}
void foo(hipsparseStatus_t err) {
  hipsparseGetErrorString(err);
  hipsparseGetErrorString(HIPSPARSE_STATUS_NOT_INITIALIZED);
}
