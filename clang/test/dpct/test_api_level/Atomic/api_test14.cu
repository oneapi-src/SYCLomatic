
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Atomic/api_test14_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Atomic/api_test14_out/MainSourceFiles.yaml | wc -l > %T/Atomic/api_test14_out/count.txt
// RUN: FileCheck --input-file %T/Atomic/api_test14_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Atomic/api_test14_out

// CHECK: 2
// TEST_FEATURE: Atomic_atomic_fetch_compare_dec

__global__ void test(int *data) {
  int inc = 1;
  atomicInc((unsigned int *)&data[5], (unsigned int)inc);
}
int main() {
  return 0;
}
