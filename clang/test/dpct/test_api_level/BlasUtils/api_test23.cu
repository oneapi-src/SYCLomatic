// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/BlasUtils/api_test23_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test23_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test23_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test23_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test23_out

// CHECK: 30


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_dot

int main() {
  hipblasHandle_t handle;
  const void *x;
  const void *y;
  void *res;

  hipblasDotEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  return 0;
}
