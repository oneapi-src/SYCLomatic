// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/BlasUtils/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test7_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test7_out

// CHECK: 24


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_getrs_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int nrhs = 275;
  int lda = 275;
  int ldb = 275;

  float **Barray_S = 0;
  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  const float **Aarray_Sc = 0;


  hipblasSgetrsBatched(handle, HIPBLAS_OP_N, n, nrhs, Aarray_Sc, lda, PivotArray, Barray_S, ldb, infoArray, batchSize);
  return 0;
}
