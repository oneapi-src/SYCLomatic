// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test9_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test9_out

// CHECK: 35


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_getri_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int lda = 275;
  int ldc = 275;

  float **Carray_S = 0;
  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;
  const float **Aarray_Sc = 0;

  hipblasSgetriBatched(handle, n, Aarray_Sc, lda, PivotArray, Carray_S, ldc, infoArray, batchSize);

  return 0;
}
