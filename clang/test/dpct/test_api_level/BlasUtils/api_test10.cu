// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test10_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test10_out

// CHECK: 37


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_geqrf_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int m = 275;
  int lda = 275;

  float **Aarray_S = 0;
  float **TauArray_S = 0;
  int *infoArray = 0;
  int batchSize = 10;

  hipblasSgeqrfBatched(handle, m, n, Aarray_S, lda, TauArray_S, infoArray, batchSize);
  return 0;
}
