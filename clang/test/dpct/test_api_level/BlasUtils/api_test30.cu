// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none    --use-custom-helper=api -out-root %T/BlasUtils/api_test30_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test30_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test30_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test30_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test30_out

// CHECK: 16


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_rot

int main() {
  hipblasHandle_t handle;
  void *x;
  void *y;
  void *sin;
  void *cos;

  hipblasRotEx(handle, 4, x, HIP_R_32F, 1,  y, HIP_R_32F, 1,  cos, sin, HIP_R_32F, HIP_R_32F);
  return 0;
}
