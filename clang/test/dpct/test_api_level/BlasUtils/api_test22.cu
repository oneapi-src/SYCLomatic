// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test22_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test22_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test22_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test22_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test22_out

// CHECK: 16


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_axpy

int main() {
  hipblasHandle_t handle;
  const void *alpha;
  const void *x;
  void *y;

  hipblasAxpyEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, y, HIP_R_32F, 1, HIP_R_32F);
  return 0;
}
