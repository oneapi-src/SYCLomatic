// RUN: dpct --format-range=none    --use-custom-helper=api -out-root %T/BlasUtils/api_test31_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test31_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test31_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test31_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test31_out

// CHECK: 15


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_scal

int main() {
  hipblasHandle_t handle;
  void * alpha;
  void * x;

  hipblasScalEx(handle, 4, alpha, HIP_R_32F, x, HIP_R_32F, 1, HIP_R_32F);
  return 0;
}
