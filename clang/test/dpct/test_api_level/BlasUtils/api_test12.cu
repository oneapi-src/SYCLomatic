// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test12_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test12_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test12_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test12_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test12_out

// CHECK: 18


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_dot

int main() {
  hipblasHandle_t handle;
  const void *x;
  const void *y;
  void *res;

  hipblasDotEx(handle, 4, x, HIP_R_32F, 1, y, HIP_R_32F, 1, res, HIP_R_32F, HIP_R_32F);
  return 0;
}
