// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test4_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test4_out

// CHECK: 40


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_getrf_batch_wrapper

int main() {
  hipblasHandle_t handle;
  int n = 275;
  int lda = 275;

  float **Aarray_S = 0;
  int *PivotArray = 0;
  int *infoArray = 0;
  int batchSize = 10;

  hipblasSgetrfBatched(handle, n, Aarray_S, lda, PivotArray, infoArray, batchSize);

  return 0;
}
