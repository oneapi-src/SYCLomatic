// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/BlasUtils/api_test25_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test25_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test25_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test25_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test25_out

// CHECK: 14


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_gemm_batch

int main() {
  hipblasHandle_t handle;
  void * alpha;
  void * beta;
  const void** a;
  const void** b;
  void** c;

  hipblasGemmBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, b, HIP_R_16F, 4, beta, c, HIP_R_16F, 4, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  return 0;
}
