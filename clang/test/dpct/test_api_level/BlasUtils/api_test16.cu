// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/BlasUtils/api_test16_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test16_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test16_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test16_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test16_out

// CHECK: 24


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_gemm_batch_stride

int main() {
  hipblasHandle_t handle;
  void * alpha;
  void * beta;
  const void * a;
  const void * b;
  void *c;

  hipblasGemmStridedBatchedEx(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, 4, 4, 4, alpha, a, HIP_R_16F, 4, 16, b, HIP_R_16F, 4, 16, beta, c, HIP_R_16F, 4, 16, 2, HIPBLAS_COMPUTE_16F, HIPBLAS_GEMM_DEFAULT);
  return 0;
}
