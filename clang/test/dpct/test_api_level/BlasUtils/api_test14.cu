// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/BlasUtils/api_test14_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/BlasUtils/api_test14_out/MainSourceFiles.yaml | wc -l > %T/BlasUtils/api_test14_out/count.txt
// RUN: FileCheck --input-file %T/BlasUtils/api_test14_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/BlasUtils/api_test14_out

// CHECK: 35


#include <hip/hip_runtime.h>
#include "hipblas.h"

// TEST_FEATURE: BlasUtils_syrk

int main() {
  hipblasHandle_t handle;
  float *alpha;
  float *beta;
  float *a;
  float *b;
  float *c;

  hipblasSsyrkx(handle, HIPBLAS_FILL_MODE_UPPER, HIPBLAS_OP_T, 2, 3, alpha, a, 3, b, 3, beta, c, 2);
  return 0;
}
