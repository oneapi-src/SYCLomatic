// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/DplExtrasMemory/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasMemory/api_test1_out/MainSourceFiles.yaml | wc -l > %T/DplExtrasMemory/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/DplExtrasMemory/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasMemory/api_test1_out

// CHECK: 14
// TEST_FEATURE: DplExtrasMemory_device_ptr

#include <thrust/device_ptr.h>

int main() {
  double *p;
  thrust::device_ptr<double> dp(p);
  return 0;
}
