// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/DplExtrasMemory/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasMemory/api_test10_out/MainSourceFiles.yaml | wc -l > %T/DplExtrasMemory/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/DplExtrasMemory/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasMemory/api_test10_out

// CHECK: 27
// TEST_FEATURE: DplExtrasMemory_get_device_pointer

#include <thrust/device_ptr.h>
int main() {
  float* a;
  thrust::device_pointer_cast<float>(a);
  return 0;
}
