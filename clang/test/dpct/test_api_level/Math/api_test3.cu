
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Math/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Math/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Math/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Math/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Math/api_test3_out

// CHECK: 2
// TEST_FEATURE: Math_length

__device__ void foo() {
  int i;
  double d;
  d = norm(i, &d);
}

int main() {
  return 0;
}
