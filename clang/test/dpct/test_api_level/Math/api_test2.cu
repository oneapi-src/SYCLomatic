
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Math/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Math/api_test2_out/MainSourceFiles.yaml | wc -l > %T/Math/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/Math/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Math/api_test2_out

// CHECK: 2
// TEST_FEATURE: Math_cast_ints_to_double

__device__ void foo() {
  int i;
  double d = __hiloint2double(i, i);
}

int main() {
  return 0;
}
