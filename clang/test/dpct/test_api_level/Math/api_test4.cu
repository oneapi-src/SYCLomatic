
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Math/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Math/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Math/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Math/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Math/api_test4_out

// CHECK: 2
// TEST_FEATURE: Math_vectorized_max

__device__ void foo() {
  unsigned u, u2;
  u = __vmaxs4(u, u2);
}

int main() {
  return 0;
}
