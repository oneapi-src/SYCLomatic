// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/FftUtils/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/FftUtils/api_test3_out/MainSourceFiles.yaml | wc -l > %T/FftUtils/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/FftUtils/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/FftUtils/api_test3_out

// CHECK: 23
// TEST_FEATURE: FftUtils_fft_engine


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

int main() {
  hipfftHandle plan;
  float2* odata;
  float2* idata;
  hipfftPlan1d(&plan, 10, HIPFFT_C2C, 3);
  hipfftExecC2C(plan, idata, odata, HIPFFT_FORWARD);
  return 0;
}
