// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/FftUtils/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/FftUtils/api_test2_out/MainSourceFiles.yaml | wc -l > %T/FftUtils/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/FftUtils/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/FftUtils/api_test2_out

// CHECK: 2
// TEST_FEATURE: FftUtils_fft_type


#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

int main() {
  hipfftType_t a = HIPFFT_C2C;
  return 0;
}
