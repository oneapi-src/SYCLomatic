
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/RngUtils/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/RngUtils/api_test3_out/MainSourceFiles.yaml | wc -l > %T/RngUtils/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/RngUtils/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/RngUtils/api_test3_out

// CHECK: 6
// TEST_FEATURE: RngUtils_create_host_rng
// TEST_FEATURE: RngUtils_typedef_host_rng_ptr

int main() {
  hiprandGenerator_t rng;
  hiprandCreateGeneratorHost(&rng, HIPRAND_RNG_PSEUDO_PHILOX4_32_10);
  return 0;
}
