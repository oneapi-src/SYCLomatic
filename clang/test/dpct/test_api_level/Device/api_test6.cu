
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test6_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test6_out

// CHECK: 37
// TEST_FEATURE: Device_device_ext_get_device_info_return_info
// TEST_FEATURE: Device_device_info_get_device_id
// TEST_FEATURE: Device_device_info_set_device_id
// TEST_FEATURE: Device_device_info_get_uuid
// TEST_FEATURE: Device_device_info_set_uuid

int main() {
  hipDeviceProp_t deviceProp;
  hipGetDeviceProperties(&deviceProp, 0);
  deviceProp.uuid;
  deviceProp.pciDeviceID;
  return 0;
}
