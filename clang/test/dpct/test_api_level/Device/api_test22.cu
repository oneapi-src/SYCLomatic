
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test22_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test22_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test22_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test22_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test22_out

// CHECK: 36
// TEST_FEATURE: Device_device_ext_get_max_compute_units

int main() {
  int res;
  hipDeviceGetAttribute(&res, hipDeviceAttributeMultiprocessorCount, 0);
  return 0;
}
