
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test7_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test7_out

// CHECK: 14
// TEST_FEATURE: Device_device_ext_is_native_atomic_supported

int main() {
  int val;
  hipDeviceGetAttribute(&val, hipDeviceAttributeComputeCapabilityMajor, 0);
  return 0;
}
