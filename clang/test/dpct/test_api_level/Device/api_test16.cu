
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test16_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test16_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test16_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test16_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test16_out

// CHECK: 16
// TEST_FEATURE: Device_dev_mgr_select_device

int main() {
  hipSetDevice(0);
  return 0;
}
