// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test31_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test31_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test31_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test31_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test31_out


// CHECK: 18

// TEST_FEATURE: Device_pointer_attributes

#include<hip/hip_runtime.h>

int main() {
  void *h_A;
  hipPointerAttribute_t attributes;
  hipPointerGetAttributes(&attributes, h_A);
  return 0;
}
