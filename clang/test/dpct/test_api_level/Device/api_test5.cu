
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test5_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test5_out

// CHECK: 17
// TEST_FEATURE: Device_device_ext_destroy_queue

int main() {
  hipStream_t s;
  hipStreamDestroy(s);
  return 0;
}
