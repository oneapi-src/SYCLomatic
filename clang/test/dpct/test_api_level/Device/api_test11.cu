
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test11_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test11_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test11_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test11_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test11_out

// CHECK: 16
// TEST_FEATURE: Device_device_ext_queues_wait_and_throw

int main() {
  hipDeviceSynchronize();
  return 0;
}
