// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/Device/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test9_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test9_out

// CHECK: 20
// TEST_FEATURE: Device_device_ext_get_saved_queue


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  float* x_S;
  int res = cublasIsamax(10, x_S, 1);
  return 0;
}
