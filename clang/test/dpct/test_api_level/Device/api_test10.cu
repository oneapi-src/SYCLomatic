// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test10_out

// CHECK: 20
// TEST_FEATURE: Device_device_ext_set_saved_queue


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipStream_t stream;
  cublasSetKernelStream(stream);
  return 0;
}
