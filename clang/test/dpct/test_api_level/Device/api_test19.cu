
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test19_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test19_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test19_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test19_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test19_out

// CHECK: 16
// TEST_FEATURE: Device_device_info_get_minor_version

int main() {
  int minor = 0;
  hipDeviceGetAttribute(&minor, hipDeviceAttributeComputeCapabilityMinor, 0);
  return 0;
}
