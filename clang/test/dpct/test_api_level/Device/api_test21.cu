
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test21_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test21_out

// CHECK: 13
// TEST_FEATURE: Device_device_ext_is_native_atomic_supported

int main() {
  int res;
  hipDeviceGetAttribute(&res, hipDeviceAttributeHostNativeAtomicSupported, 0);
  return 0;
}
