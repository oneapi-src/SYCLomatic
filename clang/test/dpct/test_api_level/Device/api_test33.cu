// RUN: dpct --format-range=none --usm-level=none  --use-custom-helper=api -out-root %T/Device/api_test33_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Device/api_test33_out/MainSourceFiles.yaml | wc -l > %T/Device/api_test33_out/count.txt
// RUN: FileCheck --input-file %T/Device/api_test33_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Device/api_test33_out

// CHECK: 15
// TEST_FEATURE: Device_device_ext_get_global_mem_size
// TEST_FEATURE: Device_device_ext_get_max_sub_group_size
// TEST_FEATURE: Device_device_ext_get_max_work_group_size
// TEST_FEATURE: Device_device_ext_get_mem_base_addr_align
// TEST_FEATURE: Device_device_ext_get_max_register_size_per_work_group
#include <hip/hip_runtime.h>
#include <hip/hip_runtime_api.h>

int main() {
  hipDevice_t device;
  int result0, result1, result2, result3, result4;
  hipDeviceGetAttribute(&result0, hipDeviceAttributeTotalConstantMemory, device);
  hipDeviceGetAttribute(&result1, hipDeviceAttributeWarpSize, device);
  hipDeviceGetAttribute(&result2, hipDeviceAttributeMaxThreadsPerBlock, device);
  hipDeviceGetAttribute(&result3, hipDeviceAttributeTextureAlignment, device);
  hipDeviceGetAttribute(&result4, hipDeviceAttributeMaxRegistersPerBlock, device);
  return 0;
}
