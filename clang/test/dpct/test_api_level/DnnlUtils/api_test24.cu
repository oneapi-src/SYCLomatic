// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test24_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test24_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test24_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test24_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test24_out

// CHECK: 29
// TEST_FEATURE: DnnlUtils_convolution_backward_bias
// TEST_FEATURE: DnnlUtils_convolution_desc

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, biasTensor;
    cudnnFilterDescriptor_t filterTensor;
    hipStream_t stream1;

    int in = 1, ic = 2, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 4, ow = 4;
    int fk = 4, fc = 1, fh = 2, fw = 2;
    int ele_num = in * ic * ih * iw;
    int oele_num = on * oc * oh * ow;
    int fele_num = fk *fc * fh * fw;
    std::vector<int> bias_dim = {1, oc, 1, 1};
    std::vector<int> bias_stride = {oc, 1, 1, 1};
    int bele_num = oc * 1;

    int filterdim[4] = {fk, fc, fh, fw};

    cudnnSetFilterNdDescriptor(filterTensor, CUDNN_DATA_FLOAT, CUDNN_TENSOR_NHWC, 4, filterdim);

    float *data, *out, *filter, *z, *bias;

    cudnnConvolutionDescriptor_t covdes;
    cudnnCreateConvolutionDescriptor(&covdes);
    cudnnSetConvolution2dDescriptor(covdes, 0, 0, 1, 1, 1, 1, CUDNN_CROSS_CORRELATION, CUDNN_DATA_FLOAT);
    cudnnSetConvolutionGroupCount(covdes, 2);
    cudnnConvolutionFwdAlgoPerf_t algo;
    int retCount = 1;

    size_t size;
    void *workspacesize;
    cudnnGetConvolutionForwardWorkspaceSize(handle, dataTensor, filterTensor, covdes, outTensor, CUDNN_CONVOLUTION_FWD_ALGO_DIRECT, &size);
    hipMalloc(&workspacesize, size);

    cudnnActivationDescriptor_t ActivationDesc;
    cudnnCreateActivationDescriptor(&ActivationDesc);
    cudnnSetActivationDescriptor(ActivationDesc, CUDNN_ACTIVATION_RELU, CUDNN_NOT_PROPAGATE_NAN, 0.0f);

    float alpha = 1.f, beta = 0.f;
    cudnnConvolutionBackwardBias(
        handle,
        &alpha,
        outTensor,
        out,
        &beta,
        biasTensor,
        bias
    );

    return 0;
}