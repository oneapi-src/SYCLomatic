// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test9_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test9_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test9_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test9_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test9_out

// CHECK: 53

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <vector>
// TEST_FEATURE: DnnlUtils_lrn_desc
// TEST_FEATURE: DnnlUtils_lrn_forward
// TEST_FEATURE: DnnlUtils_lrn_backward

int main() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    float *data, *out, *diffdata, *diffout;
    std::vector<float> host_data(ele_num);
    std::vector<float> host_out(ele_num);
    std::vector<float> host_diffdata(ele_num);
    std::vector<float> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(float));
    hipMalloc(&out, ele_num * sizeof(float));
    hipMalloc(&diffdata, ele_num * sizeof(float));
    hipMalloc(&diffout, ele_num * sizeof(float));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);

    unsigned int local_size = 3;
    float lrn_alpha = 1.5f;
    float lrn_beta = 1.5f;
    float lrn_k = 1.f;

    cudnnLRNDescriptor_t desc;
    cudnnCreateLRNDescriptor(&desc);
    cudnnSetLRNDescriptor(desc, local_size, lrn_alpha, lrn_beta, lrn_k);

    float alpha = 1.5f, beta = 0.f;
    cudnnLRNCrossChannelForward(handle, desc, CUDNN_LRN_CROSS_CHANNEL_DIM1, &alpha, dataTensor, data, &beta, outTensor, out);
    //cudaMemcpy(host_out.data(), out, ele_num * sizeof(float), cudaMemcpyDeviceToHost);
    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    auto s = cudnnLRNCrossChannelBackward(handle, desc, CUDNN_LRN_CROSS_CHANNEL_DIM1, &alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();
    //check(s);
    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
    return 0;
}
