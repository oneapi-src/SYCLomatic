// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test14_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test14_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test14_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test14_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test14_out


// CHECK: 13
// TEST_FEATURE: DnnlUtils_batch_normalization_forward_training
// TEST_FEATURE: DnnlUtils_batch_normalization_mode
// TEST_FEATURE: DnnlUtils_batch_normalization_ops
// TEST_FEATURE: DnnlUtils_get_batch_normalization_workspace_size
#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <iostream>
#include <vector>

int main() {
    int nDevices;
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, scalebiasTensor, additionTensor;
    hipStream_t stream1;

    int in = 2, ic = 4, ih = 5, iw = 5;
    int on = 2, oc = 4, oh = 5, ow = 5;
    int sbn = 1, sbc = 4, sbh = 5, sbw = 5;
    int ele_num = in* ic * ih * iw;
    int oele_num = on* oc * oh * ow;
    int sele_num = sbn*sbc * sbh * sbw;

    int save = 1;
    float *data, *out, *scale, *bias, *rmean, *rvar, *smean, *svar, *z;

    float alpha = 1.f, beta = 0.f, eps = 1.f;
    double factor = 0.1f;
    auto status = cudnnBatchNormalizationForwardTraining(
        handle,
        CUDNN_BATCHNORM_PER_ACTIVATION,
        &alpha,
        &beta,
        dataTensor,
        data,
        outTensor,
        out,
        scalebiasTensor,
        scale,
        bias,
        factor,
        rmean,
        rvar,
        eps,
        smean,
        svar);

    return 0;
}