// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test7_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test7_out

// CHECK: 53

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <vector>
// TEST_FEATURE: DnnlUtils_activation_desc
// TEST_FEATURE: DnnlUtils_activation_forward
// TEST_FEATURE: DnnlUtils_activation_backward

int main() {

    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    //using float = dt_trait<T>::type;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    float *data, *out, *diffdata, *diffout;
    std::vector<float> host_data(ele_num);
    std::vector<float> host_out(ele_num);
    std::vector<float> host_diffdata(ele_num);
    std::vector<float> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(float));
    hipMalloc(&out, ele_num * sizeof(float));
    hipMalloc(&diffdata, ele_num * sizeof(float));
    hipMalloc(&diffout, ele_num * sizeof(float));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);

    cudnnActivationDescriptor_t desc;
    cudnnCreateActivationDescriptor(&desc);
    cudnnSetActivationDescriptor(desc, CUDNN_ACTIVATION_SIGMOID, CUDNN_PROPAGATE_NAN, 0.f);

    float alpha = 1.5f, beta = 0.f;
    cudnnActivationForward(handle, desc, &alpha, dataTensor, data, &beta, outTensor, out);

    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    cudnnActivationBackward(handle, desc, &alpha, outTensor, out, diffoutTensor, diffout, dataTensor, data, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();

    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);

    return 0;
}
