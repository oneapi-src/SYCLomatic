// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/DnnlUtils/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/DnnlUtils/api_test10_out/MainSourceFiles.yaml | wc -l > %T/DnnlUtils/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/DnnlUtils/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DnnlUtils/api_test10_out

// CHECK: 53

#include <hip/hip_runtime.h>
#include <cudnn.h>
#include <vector>
// TEST_FEATURE: DnnlUtils_softmax_algorithm
// TEST_FEATURE: DnnlUtils_softmax_mode
// TEST_FEATURE: DnnlUtils_softmax_forward
// TEST_FEATURE: DnnlUtils_softmax_backward

int main() {
    cudnnHandle_t handle;
    cudnnTensorDescriptor_t dataTensor, outTensor, diffdataTensor, diffoutTensor;

    cudnnCreate(&handle);

    hipStream_t stream1;
    hipStreamCreate(&stream1);
    cudnnSetStream(handle, stream1);

    cudnnCreateTensorDescriptor(&dataTensor);
    cudnnCreateTensorDescriptor(&outTensor);
    cudnnCreateTensorDescriptor(&diffdataTensor);
    cudnnCreateTensorDescriptor(&diffoutTensor);
    int n = 1, c = 2, h = 5, w = 5;
    int ele_num = n * c * h * w;

    //using float = dt_trait<T>::type;

    cudnnSetTensor4dDescriptor(dataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(outTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffdataTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    cudnnSetTensor4dDescriptor(diffoutTensor, CUDNN_TENSOR_NCHW, CUDNN_DATA_FLOAT, n, c, h, w);
    float *data, *out, *diffdata, *diffout;
    std::vector<float> host_data(ele_num);
    std::vector<float> host_out(ele_num);
    std::vector<float> host_diffdata(ele_num);
    std::vector<float> host_diffout(ele_num);
    for(int i = 0; i < ele_num; i++) {
        host_data[i] = i * 0.1f;
        host_out[i] = i;
        host_diffdata[i] = i;
        host_diffout[i] = 1.f;
    }

    hipMalloc(&data, ele_num * sizeof(float));
    hipMalloc(&out, ele_num * sizeof(float));
    hipMalloc(&diffdata, ele_num * sizeof(float));
    hipMalloc(&diffout, ele_num * sizeof(float));

    hipMemcpy(data, host_data.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(out, host_out.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffdata, host_diffdata.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);
    hipMemcpy(diffout, host_diffout.data(), ele_num * sizeof(float), hipMemcpyHostToDevice);

    float alpha = 1.5f, beta = 0.f;
    cudnnSoftmaxForward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, dataTensor, data, &beta, outTensor, out);
    hipMemcpy(host_out.data(), out, ele_num * sizeof(float), hipMemcpyDeviceToHost);
    alpha = 2.f, beta = 0.f;
    hipDeviceSynchronize();
    cudnnSoftmaxBackward(handle, CUDNN_SOFTMAX_ACCURATE, CUDNN_SOFTMAX_MODE_CHANNEL, &alpha, outTensor, out, diffoutTensor, diffout, &beta, diffdataTensor, diffdata);
    hipDeviceSynchronize();

    hipMemcpy(host_diffdata.data(), diffdata, ele_num * sizeof(float), hipMemcpyDeviceToHost);
    hipDeviceSynchronize();


    cudnnDestroy(handle);
    hipFree(data);
    hipFree(out);
    hipFree(diffdata);
    hipFree(diffout);
    return 0;
}
