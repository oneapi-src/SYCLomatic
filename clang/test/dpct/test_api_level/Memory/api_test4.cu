
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test4_out

// CHECK: 26
// TEST_FEATURE: Memory_dpct_malloc

int main() {
  float* a;
  hipMalloc(&a, 4);
  return 0;
}
