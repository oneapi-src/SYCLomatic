
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test19_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test19_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test19_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test19_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test19_out

// CHECK: 45
// TEST_FEATURE: Memory_shared_memory_alias

__managed__ float A[1024];

int main() {
  return 0;
}
