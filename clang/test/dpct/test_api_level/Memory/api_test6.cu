
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test6_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test6_out

// CHECK: 26
// TEST_FEATURE: Memory_dpct_free

int main() {
  float* a;
  hipFree(a);
  return 0;
}
