
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test16_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test16_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test16_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test16_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test16_out

// CHECK: 50
// TEST_FEATURE: Memory_constant_memory_alias

__constant__ float A[1024];

int main() {
  return 0;
}
