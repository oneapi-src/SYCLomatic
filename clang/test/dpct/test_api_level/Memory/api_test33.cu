
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test33_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test33_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test33_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test33_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test33_out

// CHECK: 29
// TEST_FEATURE: Memory_get_access

__global__ void foo(float* f) {
}

int main() {
  float* f;
  hipMalloc(&f, 8);
  foo<<<1, 1>>>(f);
  return 0;
}
