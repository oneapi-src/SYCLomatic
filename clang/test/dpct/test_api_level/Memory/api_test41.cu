
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test41_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test41_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test41_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test41_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test41_out

// CHECK: 29
// TEST_FEATURE: Memory_dpct_memset
// TEST_FEATURE: Memory_dpct_memset_2d
// TEST_FEATURE: Memory_dpct_memset_3d

int main() {
  hipExtent e = make_hipExtent(1, 1, 1);
  hipPitchedPtr p_A;
  hipMemset3D(p_A, 0xf, e);
  return 0;
}
