
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test17_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test17_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test17_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test17_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test17_out

// CHECK: 44
// TEST_FEATURE: Memory_constant_memory_alias

__constant__ float A[1024];

int main() {
  return 0;
}
