
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test21_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test21_out

// CHECK: 46
// TEST_FEATURE: Memory_device_memory_get_size

static __device__ float d_A[1234567];

int main() {
  size_t size2;
  hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));
  return 0;
}
