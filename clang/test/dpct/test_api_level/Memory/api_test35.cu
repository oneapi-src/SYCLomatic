
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test35_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test35_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test35_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test35_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test35_out

// CHECK: 32
// TEST_FEATURE: Memory_dpct_memcpy

int main() {
  float constData[1234567 * 4];
  float* h_A;
  int size;
  hipMemcpyToSymbol(HIP_SYMBOL(constData), h_A, size, 1, hipMemcpyHostToDevice);
  return 0;
}
