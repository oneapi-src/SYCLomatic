
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test42_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test42_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test42_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test42_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test42_out

// CHECK: 37
// TEST_FEATURE: Memory_async_dpct_memset
// TEST_FEATURE: Memory_async_dpct_memset_2d
// TEST_FEATURE: Memory_async_dpct_memset_3d

int main() {
  hipExtent e = make_hipExtent(1, 1, 1);
  hipPitchedPtr p_A;
  hipMemset3DAsync(p_A, 0xf, e);
  return 0;
}
