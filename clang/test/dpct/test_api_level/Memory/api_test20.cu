
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test20_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test20_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test20_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test20_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test20_out

// CHECK: 52
// TEST_FEATURE: Memory_device_memory_get_size

static __device__ float d_A[1234567];

int main() {
  size_t size2;
  hipGetSymbolSize(&size2, HIP_SYMBOL(d_A));
  return 0;
}
