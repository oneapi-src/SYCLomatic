
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test3_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test3_out

// CHECK: 25
// TEST_FEATURE: Memory_dpct_malloc_3d
// TEST_FEATURE: Memory_dpct_malloc_2d

int main() {
  hipExtent extent = make_hipExtent(1, 1, 1);
  hipPitchedPtr p3;
  hipMalloc3D(&p3, extent);
  return 0;
}
