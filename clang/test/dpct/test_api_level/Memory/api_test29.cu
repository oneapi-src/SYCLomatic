
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test29_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test29_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test29_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test29_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test29_out

// CHECK: 54
// TEST_FEATURE: Memory_device_memory_get_ptr_q

__constant__ float constData[4];

int main() {
  hipStream_t stream;
  float* host;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(constData), host, 1, 3, hipMemcpyHostToDevice, stream);
  return 0;
}
