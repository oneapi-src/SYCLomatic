
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none   --use-custom-helper=api -out-root %T/Memory/api_test39_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test39_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test39_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test39_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test39_out

// CHECK: 36
// TEST_FEATURE: Memory_async_dpct_memcpy
// TEST_FEATURE: Memory_async_dpct_memcpy_2d
// TEST_FEATURE: Memory_async_dpct_memcpy_3d

int main() {
  hipMemcpy3DParms parms;
  hipMemcpy3DAsync(&parms);
  return 0;
}
