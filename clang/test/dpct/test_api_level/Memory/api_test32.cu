
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Memory/api_test32_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test32_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test32_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test32_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test32_out

// CHECK: 30
// TEST_FEATURE: Memory_access_wrapper

__global__ void foo(float* f) {
}

int main() {
  float* f;
  hipMalloc(&f, 8);
  f = f + 1;
  foo<<<1, 1>>>(f);
  return 0;
}