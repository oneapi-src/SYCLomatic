
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Memory/api_test26_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Memory/api_test26_out/MainSourceFiles.yaml | wc -l > %T/Memory/api_test26_out/count.txt
// RUN: FileCheck --input-file %T/Memory/api_test26_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Memory/api_test26_out

// CHECK: 48
// TEST_FEATURE: Memory_device_memory_get_access
// TEST_FEATURE: Memory_device_memory_init_q
// TEST_FEATURE: Memory_dpct_accessor

__device__ float c[16][16];

__global__ void kernel() {
  c[0][0] = 1.0f;
}

int main() {
  hipStream_t s;
  kernel<<<1, 1, 0, s>>>();
  return 0;
}
