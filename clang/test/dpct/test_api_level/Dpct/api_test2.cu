
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none --sycl-named-lambda  --use-custom-helper=api -out-root %T/Dpct/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Dpct/api_test2_out/MainSourceFiles.yaml | wc -l > %T/Dpct/api_test2_out/count.txt
// RUN: cat %T/Dpct/api_test2_out/include/dpct/dpct.hpp >> %T/Dpct/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/Dpct/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Dpct/api_test2_out

// CHECK: 19

// CHECK: template <class... Args> class dpct_kernel_name;
// CHECK-NEXT: template <int Arg> class dpct_kernel_scalar;

// TEST_FEATURE: Dpct_dpct_named_lambda

__global__ void foo() {}

int main() {
  foo<<<1, 1>>>();
  return 0;
}
