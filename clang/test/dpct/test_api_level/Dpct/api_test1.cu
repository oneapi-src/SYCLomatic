
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Dpct/api_test1_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Dpct/api_test1_out/MainSourceFiles.yaml | wc -l > %T/Dpct/api_test1_out/count.txt
// RUN: FileCheck --input-file %T/Dpct/api_test1_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Dpct/api_test1_out

// CHECK: 31

// TEST_FEATURE: Dpct_non_local_include_dependency
// TEST_FEATURE: Dpct_dpct_align_and_inline
// TEST_FEATURE: Dpct_dpct_noinline
// TEST_FEATURE: Dpct_check_error_code

class __align__(8) T1 {
    unsigned int l, a;
};

__forceinline__ void foo(){}

__noinline__ hipError_t foo2(){
  void **buffer;
  size_t size;
  return hipMalloc(buffer, size);
}

int main() {
  return 0;
}
