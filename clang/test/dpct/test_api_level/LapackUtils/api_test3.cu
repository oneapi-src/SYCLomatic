// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test3_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test3_out

// CHECK: 22
// TEST_FEATURE: LapackUtils_potrf_batch


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  float ** a_s_ptrs;
  int *infoArray;
  hipsolverDnSpotrfBatched(handle, HIPBLAS_FILL_MODE_UPPER, 3, a_s_ptrs, 3, infoArray, 2);
  return 0;
}
