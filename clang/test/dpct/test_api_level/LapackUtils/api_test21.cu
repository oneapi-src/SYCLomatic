// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test21_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test21_out

// CHECK: 30
// TEST_FEATURE: LapackUtils_syhegvd_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipblasFillMode_t uplo;
  int n;
  const float *A;
  int lda;
  const float *B;
  int ldb;
  const float *W;
  int *lwork;
  hipsolverSyevjInfo_t params;

  hipsolverDnSsygvj_bufferSize(handle, itype, jobz, uplo, n, A, lda, B, ldb, W,
                              lwork, params);
  return 0;
}
