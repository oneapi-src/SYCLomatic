// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test21_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test21_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test21_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test21_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test21_out

// CHECK: 31
// TEST_FEATURE: LapackUtils_syhegvd_scratchpad_size


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipblasFillMode_t uplo;
  int n;
  const float *A;
  int lda;
  const float *B;
  int ldb;
  const float *W;
  int *lwork;
  hipsolverSyevjInfo_t params;

  hipsolverDnSsygvj_bufferSize(handle, itype, jobz, uplo, n, A, lda, B, ldb, W,
                              lwork, params);
  return 0;
}
