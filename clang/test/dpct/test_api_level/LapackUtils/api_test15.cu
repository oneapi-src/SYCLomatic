// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test15_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test15_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test15_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test15_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test15_out

// CHECK: 35
// TEST_FEATURE: LapackUtils_syheevx_scratchpad_size_T


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigMode_t jobz;
  hipsolverEigRange_t range;
  hipblasFillMode_t uplo;
  int n;
  const float *A;
  int lda;
  float vl;
  float vu;
  int il;
  int iu;
  int *h_meig;
  const float *W;
  int *lwork;

  hipsolverDnSsyevdx_bufferSize(handle, jobz, range, uplo, n, A, lda, vl, vu, il,
                               iu, h_meig, W, lwork);
  return 0;
}
