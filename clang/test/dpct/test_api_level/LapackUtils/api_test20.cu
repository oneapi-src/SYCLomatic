// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test20_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test20_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test20_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test20_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test20_out

// CHECK: 38
// TEST_FEATURE: LapackUtils_syhegvx


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  hipsolverHandle_t handle;
  hipsolverEigType_t itype;
  hipsolverEigMode_t jobz;
  hipsolverEigRange_t range;
  hipblasFillMode_t uplo;
  int n;
  float *A;
  int lda;
  float *B;
  int ldb;
  float vl;
  float vu;
  int il;
  int iu;
  int *h_meig;
  float *W;
  float *work;
  int lwork;
  int *devInfo;

  hipsolverDnSsygvdx(handle, itype, jobz, range, uplo, n, A, lda, B, ldb, vl, vu,
                    il, iu, h_meig, W, work, lwork, devInfo);
  return 0;
}
