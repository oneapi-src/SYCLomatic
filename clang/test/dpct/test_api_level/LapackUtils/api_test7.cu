// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2
// RUN: dpct --format-range=none --use-custom-helper=api -out-root %T/LapackUtils/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/LapackUtils/api_test7_out/MainSourceFiles.yaml | wc -l > %T/LapackUtils/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/LapackUtils/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/LapackUtils/api_test7_out

// CHECK: 34
// TEST_FEATURE: LapackUtils_getrs


#include <hip/hip_runtime.h>
#include "hipsolver.h"

int main() {
  float* a_s;
  int64_t* ipiv_s;
  float* b_s;
  hipsolverHandle_t handle;
  hipsolverDnParams_t params;
  int *info;

  hipsolverDnXgetrs(handle, params, HIPBLAS_OP_N, 2, 3, HIP_R_32F, a_s, 2, ipiv_s, HIP_R_32F, b_s, 2, info);
  return 0;
}
