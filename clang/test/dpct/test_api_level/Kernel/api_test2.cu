
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Kernel/api_test2_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Kernel/api_test2_out/MainSourceFiles.yaml | wc -l > %T/Kernel/api_test2_out/count.txt
// RUN: FileCheck --input-file %T/Kernel/api_test2_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Kernel/api_test2_out

// CHECK: 16
// TEST_FEATURE: Kernel_get_kernel_function_info

__global__ void foo() {}

int main() {
  hipFuncAttributes attrs;
  hipFuncGetAttributes(&attrs, reinterpret_cast<const void*>(foo));
  return 0;
}
