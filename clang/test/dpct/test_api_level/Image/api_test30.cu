// UNSUPPORTED: cuda-12.0
// UNSUPPORTED: v12.0
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test30_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test30_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test30_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test30_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test30_out

// CHECK: 42
// TEST_FEATURE: Image_image_channel

int main() {
  texture<unsigned int, 1, hipReadModeElementType> tex_tmp;
  hipTextureAddressMode addr = tex_tmp.addressMode[0];
  return 0;
}
