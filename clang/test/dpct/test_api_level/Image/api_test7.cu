// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test7_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test7_out

// CHECK: 2
// TEST_FEATURE: Image_image_data_type

#include "hip/hip_runtime.h"

int main() {
  HIPresourcetype_enum a;
  return 0;
}
