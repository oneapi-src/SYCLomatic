// UNSUPPORTED: cuda-12.0, cuda-12.1
// UNSUPPORTED: v12.0, v12.1
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test26_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test26_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test26_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test26_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test26_out

// CHECK: 52
// TEST_FEATURE: Image_sampling_info_set_addressing_mode_filtering_mode_is_normalized

int main() {
  texture<unsigned int, 1, hipReadModeElementType> tex_tmp;
  if (true) {
    tex_tmp.filterMode = hipFilterModePoint;
    tex_tmp.addressMode[1] = hipAddressModeWrap;
    tex_tmp.addressMode[2] = hipAddressModeWrap;
    tex_tmp.normalized = 1;
  }
  return 0;
}
