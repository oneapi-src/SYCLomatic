// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test17_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test17_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test17_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test17_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test17_out

// CHECK: 28
// TEST_FEATURE: Image_image_wrapper_base_set_filtering_mode

#include "hip/hip_runtime.h"

int main() {
  hipTexRef tex;
  HIPfilter_mode filter_mode;
  hipTexRefSetFilterMode(tex, filter_mode);
  return 0;
}
