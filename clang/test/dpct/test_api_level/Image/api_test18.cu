// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Image/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Image/api_test18_out/MainSourceFiles.yaml | wc -l > %T/Image/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/Image/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Image/api_test18_out

// CHECK: 30
// TEST_FEATURE: Image_image_wrapper_base_set_channel_type
// TEST_FEATURE: Image_image_wrapper_base_set_channel_num

#include "hip/hip_runtime.h"

int main() {
  hipTexRef tex;
  hipArray_Format format;
  hipTexRefSetFormat(tex, format, 4);
  return 0;
}
