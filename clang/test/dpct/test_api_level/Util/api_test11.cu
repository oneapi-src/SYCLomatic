// RUN: dpct --format-range=none  --use-custom-helper=api -out-root %T/Util/api_test11_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test11_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test11_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test11_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test11_out

// CHECK: 34

// TEST_FEATURE: Util_matrix_mem_copy_T


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  hipblasHandle_t handle;
  float* a;
  float *alpha;
  hipblasStrmm(handle, HIPBLAS_SIDE_LEFT, HIPBLAS_FILL_MODE_LOWER, HIPBLAS_OP_N, HIPBLAS_DIAG_NON_UNIT, 4, 4, alpha, a, 4, a, 4, a, 4);
  return 0;
}
