
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test4_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test4_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test4_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test4_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test4_out

// CHECK: 2
// TEST_FEATURE: Util_vectorized_max

__device__ void foo() {
  unsigned u, u2;
  u = __vmaxs4(u, u2);
}

int main() {
  return 0;
}
