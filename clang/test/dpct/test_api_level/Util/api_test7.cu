
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test7_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test7_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test7_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test7_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test7_out

// CHECK: 2
// TEST_FEATURE: Util_reverse_bits

__device__ void foo() {
  unsigned u;
  u = __brev(u);
}

int main() {
  return 0;
}
