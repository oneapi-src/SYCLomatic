
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test8_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test8_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test8_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test8_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test8_out

// CHECK: 2
// TEST_FEATURE: Util_byte_level_permute

__device__ void foo() {
  unsigned u;
  u = __byte_perm(u, u, u);
}

int main() {
  return 0;
}
