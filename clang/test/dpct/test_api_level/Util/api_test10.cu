// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test10_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test10_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test10_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test10_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test10_out

// CHECK: 34

// TEST_FEATURE: Util_matrix_mem_copy


#include <hip/hip_runtime.h>
#include "hipblas.h"

int main() {
  float* a;
  hipblasSetVector(10, sizeof(float), a, 1, a, 1);
  return 0;
}
