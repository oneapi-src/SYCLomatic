#include "hip/hip_runtime.h"
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test18_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test18_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test18_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test18_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test18_out

// CHECK: 1
// TEST_FEATURE: Util_cdiv

#include <hip/hip_complex.h>

__device__ void foo1() {
    float2 a, b;

    auto c = hipCdiv(a, b);

}

int main() {
    return 0;
}