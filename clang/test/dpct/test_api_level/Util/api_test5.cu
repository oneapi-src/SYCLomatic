
#include <hip/hip_runtime.h>
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/Util/api_test5_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only
// RUN: grep "IsCalled" %T/Util/api_test5_out/MainSourceFiles.yaml | wc -l > %T/Util/api_test5_out/count.txt
// RUN: FileCheck --input-file %T/Util/api_test5_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/Util/api_test5_out

// CHECK: 2
// TEST_FEATURE: Util_vectorized_min

__device__ void foo() {
  unsigned u, u2;
  u = __vminu2(u, u2);
  u = __vminu4(u, u2);
}

int main() {
  return 0;
}
