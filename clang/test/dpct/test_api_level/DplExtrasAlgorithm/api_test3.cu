// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/DplExtrasAlgorithm/api_test3_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasAlgorithm/api_test3_out/MainSourceFiles.yaml | wc -l > %T/DplExtrasAlgorithm/api_test3_out/count.txt
// RUN: FileCheck --input-file %T/DplExtrasAlgorithm/api_test3_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasAlgorithm/api_test3_out

// CHECK: 37
// TEST_FEATURE: DplExtrasAlgorithm_remove_copy_if

#include <thrust/remove.h>

struct greater_than_zero
{
  __host__ __device__
  bool operator()(int x) const
  {
    return x > 0;
  }
  typedef int argument_type;
};

int main() {
  int A[10];
  greater_than_zero pred;
  thrust::remove_copy_if(A, A + 10, A, A, pred);
  return 0;
}
