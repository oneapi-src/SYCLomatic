// UNSUPPORTED: cuda-8.0
// UNSUPPORTED: v8.0
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/DplExtrasAlgorithm/api_test6_out %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasAlgorithm/api_test6_out/MainSourceFiles.yaml | wc -l > %T/DplExtrasAlgorithm/api_test6_out/count.txt
// RUN: FileCheck --input-file %T/DplExtrasAlgorithm/api_test6_out/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasAlgorithm/api_test6_out

// CHECK: 38
// TEST_FEATURE: DplExtrasAlgorithm_stable_sort

#include <thrust/sort.h>
int main() {
  int *a;
  thrust::stable_sort_by_key(a, a + 10, a);
  return 0;
}
