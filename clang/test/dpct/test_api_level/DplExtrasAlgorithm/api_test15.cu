// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.2
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.2
// RUN: dpct --format-range=none  --usm-level=none  --use-custom-helper=api -out-root %T/DplExtrasAlgorithm/api_test15 %s --cuda-include-path="%cuda-path/include" -- -x cuda --cuda-host-only -std=c++17
// RUN: grep "IsCalled" %T/DplExtrasAlgorithm/api_test15/MainSourceFiles.yaml | wc -l > %T/DplExtrasAlgorithm/api_test15/count.txt
// RUN: FileCheck --input-file %T/DplExtrasAlgorithm/api_test15/count.txt --match-full-lines %s
// RUN: rm -rf %T/DplExtrasAlgorithm/api_test15

// CHECK: 32
// TEST_FEATURE: DplExtrasAlgorithm_segmented_sort_pairs

#include <hipcub/hipcub.hpp>

int main() {
   void *temp_storage;
  size_t temp_storage_size;
  int n, num_segments, *d_keys_in, *d_keys_out, *d_values_in, *d_values_out, *d_offsets;
  hipcub::DeviceSegmentedRadixSort::SortPairs(temp_storage, temp_storage_size, d_keys_in, d_keys_out, d_values_in, d_values_out, n, num_segments, d_offsets, d_offsets + 1);
  return 0;
}
