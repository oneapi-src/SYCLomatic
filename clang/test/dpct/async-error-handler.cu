
#include <hip/hip_runtime.h>
// RUN: dpct --usm-level=none -out-root %T/async-error-handler %s --cuda-include-path="%cuda-path/include" --sycl-named-lambda --always-use-async-handler -- -std=c++14 -x cuda --cuda-host-only
// RUN: FileCheck --input-file %T/async-error-handler/async-error-handler.dp.cpp --match-full-lines %s


int main() {
  // CHECK: dpct::device_ext &dev_ct1 = dpct::get_current_device();
  // CHECK: dpct::queue_ptr s0, s1, s2;
  hipStream_t s0, s1, s2;

  // CHECK: s0 = dev_ct1.create_queue(true);
  hipStreamCreate(&s0);

  // CHECK: /*
  // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag and priority options.
  // CHECK-NEXT: */
  // CHECK-NEXT: s1 = dev_ct1.create_queue(true);
  hipStreamCreateWithFlags(&s1, hipStreamDefault);

  // CHECK: /*
  // CHECK-NEXT: DPCT1025:{{[0-9]+}}: The SYCL queue is created ignoring the flag and priority options.
  // CHECK-NEXT: */
  // CHECK-NEXT: s2 = dev_ct1.create_queue(true);
  hipStreamCreateWithPriority(&s2, hipStreamDefault, 2);
}

