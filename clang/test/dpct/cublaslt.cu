// UNSUPPORTED: cuda-8.0, cuda-9.0, cuda-9.1, cuda-9.2, cuda-10.0, cuda-10.1, cuda-10.2, cuda-11.0, cuda-11.1, cuda-11.2, cuda-11.3, cuda-11.4, cuda-11.5, cuda-11.6, cuda-11.7, cuda-11.8, cuda-12.0, cuda-12.1, cuda-12.2, cuda-12.3
// UNSUPPORTED: v8.0, v9.0, v9.1, v9.2, v10.0, v10.1, v10.2, v11.0, v11.1, v11.2, v11.3, v11.4, v11.5, v11.6, v11.7, v11.8, v12.0, v12.1, v12.2, v12.3
// RUN: dpct --format-range=none --out-root %T/cublaslt %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublaslt/cublaslt.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublaslt/cublaslt.dp.cpp -o %T/cublaslt/cublaslt.dp.o %}


#include <hip/hip_runtime.h>
#include "hipblaslt.h"

void foo1 () {
  // CHECK: dpct::blas_gemm::experimental::descriptor_ptr ltHandle;
  // CHECK-NEXT: ltHandle = new dpct::blas_gemm::experimental::descriptor();
  // CHECK-NEXT: delete (ltHandle);
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  hipblasLtDestroy(ltHandle);

  // CHECK: dpct::blas_gemm::experimental::matrix_layout_ptr matLayout;
  // CHECK-NEXT: dpct::library_data_t type;
  // CHECK-NEXT: uint64_t rows;
  // CHECK-NEXT: uint64_t cols;
  // CHECK-NEXT: int64_t ld;
  // CHECK-NEXT: matLayout = new dpct::blas_gemm::experimental::matrix_layout_t(type, rows, cols, ld);
  hipblasLtMatrixLayout_t matLayout;
  hipDataType type;
  uint64_t rows;
  uint64_t cols;
  int64_t ld;
  hipblasLtMatrixLayoutCreate(&matLayout, type, rows, cols, ld);

  // CHECK: dpct::blas_gemm::experimental::matrix_layout_t::attribute attr1;
  // CHECK-NEXT: void *buf1;
  // CHECK-NEXT: size_t sizeInBytes1;
  // CHECK-NEXT: size_t *sizeWritten1;
  // CHECK-NEXT: matLayout->get_attribute(attr1, buf1);
  // CHECK-NEXT: matLayout->set_attribute(attr1, buf1);
  // CHECK-NEXT: delete (matLayout);
  hipblasLtMatrixLayoutAttribute_t attr1;
  void *buf1;
  size_t sizeInBytes1;
  size_t *sizeWritten1;
  hipblasLtMatrixLayoutGetAttribute(matLayout, attr1, buf1, sizeInBytes1, sizeWritten1);
  hipblasLtMatrixLayoutSetAttribute(matLayout, attr1, buf1, sizeInBytes1);
  hipblasLtMatrixLayoutDestroy(matLayout);

  // CHECK: dpct::blas_gemm::experimental::matmul_desc_ptr matmulDesc;
  // CHECK-NEXT: dpct::compute_type computeType;
  // CHECK-NEXT: dpct::library_data_t scaleType;
  // CHECK-NEXT: matmulDesc = new dpct::blas_gemm::experimental::matmul_desc_t(computeType, scaleType);
  hipblasLtMatmulDesc_t matmulDesc;
  hipblasComputeType_t computeType;
  hipDataType scaleType;
  hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType);

  // CHECK: dpct::blas_gemm::experimental::matmul_desc_t::attribute attr2;
  // CHECK-NEXT: void *buf2;
  // CHECK-NEXT: size_t sizeInBytes2;
  // CHECK-NEXT: size_t *sizeWritten2;
  // CHECK-NEXT: matmulDesc->get_attribute(attr2, buf2);
  // CHECK-NEXT: matmulDesc->set_attribute(attr2, buf2);
  // CHECK-NEXT: delete (matmulDesc);
  hipblasLtMatmulDescAttributes_t attr2;
  void *buf2;
  size_t sizeInBytes2;
  size_t *sizeWritten2;
  hipblasLtMatmulDescGetAttribute(matmulDesc, attr2, buf2, sizeInBytes2, sizeWritten2);
  hipblasLtMatmulDescSetAttribute(matmulDesc, attr2, buf2, sizeInBytes2);
  hipblasLtMatmulDescDestroy(matmulDesc);

  // CHECK: int matmulPreference;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasLtMatmulPreferenceCreate was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: void *buf3;
  // CHECK-NEXT: size_t sizeInBytes3;
  // CHECK-NEXT: size_t *sizeWritten3;
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasLtMatmulPreferenceGetAttribute was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasLtMatmulPreferenceSetAttribute was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  // CHECK-NEXT: /*
  // CHECK-NEXT: DPCT1026:{{[0-9]+}}: The call to cublasLtMatmulPreferenceDestroy was removed because this functionality is redundant in SYCL.
  // CHECK-NEXT: */
  hipblasLtMatmulPreference_t matmulPreference;
  hipblasLtMatmulPreferenceCreate(&matmulPreference);
  void *buf3;
  size_t sizeInBytes3;
  size_t *sizeWritten3;
  hipblasLtMatmulPreferenceGetAttribute(matmulPreference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, buf3, sizeInBytes3, sizeWritten3);
  hipblasLtMatmulPreferenceSetAttribute(matmulPreference, HIPBLASLT_MATMUL_PREF_MAX_WORKSPACE_BYTES, buf3, sizeInBytes3);
  hipblasLtMatmulPreferenceDestroy(matmulPreference);

  hipblasLtMatrixLayout_t Adesc;
  hipblasLtMatrixLayout_t Bdesc;
  hipblasLtMatrixLayout_t Cdesc;
  hipblasLtMatrixLayout_t Ddesc;

  // CHECK: int requestedAlgoCount = 1;
  // CHECK-NEXT: int heuristicResultsArray;
  // CHECK-NEXT: int returnAlgoCount;
  // CHECK-NEXT: returnAlgoCount = 1;
  int requestedAlgoCount = 1;
  hipblasLtMatmulHeuristicResult_t heuristicResultsArray;
  int returnAlgoCount;
  hipblasLtMatmulAlgoGetHeuristic(ltHandle, matmulDesc, Adesc, Bdesc, Cdesc, Ddesc, matmulPreference, requestedAlgoCount, &heuristicResultsArray, &returnAlgoCount);
}

void foo2() {
  // CHECK: dpct::blas_gemm::experimental::descriptor_ptr lightHandle;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matmul_desc_ptr computeDesc;
  // CHECK-NEXT: const void *alpha;
  // CHECK-NEXT: const void *A;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Adesc;
  // CHECK-NEXT: const void *B;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Bdesc;
  // CHECK-NEXT: const void *beta;
  // CHECK-NEXT: const void *C;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Cdesc;
  // CHECK-NEXT: void *D;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Ddesc;
  // CHECK-NEXT: const int *algo;
  // CHECK-NEXT: void *workspace;
  // CHECK-NEXT: size_t workspaceSizeInBytes;
  // CHECK-NEXT: dpct::queue_ptr stream;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matmul(lightHandle, computeDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, stream);
  hipblasLtHandle_t lightHandle;
  hipblasLtMatmulDesc_t computeDesc;
  const void *alpha;
  const void *A;
  hipblasLtMatrixLayout_t Adesc;
  const void *B;
  hipblasLtMatrixLayout_t Bdesc;
  const void *beta;
  const void *C;
  hipblasLtMatrixLayout_t Cdesc;
  void *D;
  hipblasLtMatrixLayout_t Ddesc;
  const hipblasLtMatmulAlgo_t *algo;
  void *workspace;
  size_t workspaceSizeInBytes;
  hipStream_t stream;
  hipblasLtMatmul(lightHandle, computeDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, algo, workspace, workspaceSizeInBytes, stream);
}

void foo3() {
  // CHECK: dpct::blas_gemm::experimental::order_t a;
  // CHECK-NEXT: a = dpct::blas_gemm::experimental::order_t::col;
  // CHECK-NEXT: a = dpct::blas_gemm::experimental::order_t::row;
  // CHECK-NEXT: a = dpct::blas_gemm::experimental::order_t::col32;
  // CHECK-NEXT: a = dpct::blas_gemm::experimental::order_t::col4_4r2_8c;
  // CHECK-NEXT: a = dpct::blas_gemm::experimental::order_t::col32_2r_4r4;
  hipblasLtOrder_t a;
  a = HIPBLASLT_ORDER_COL;
  a = HIPBLASLT_ORDER_ROW;
  a = CUBLASLT_ORDER_COL32;
  a = CUBLASLT_ORDER_COL4_4R2_8C;
  a = CUBLASLT_ORDER_COL32_2R_4R4;
  // CHECK: dpct::blas_gemm::experimental::pointer_mode_t b;
  // CHECK-NEXT: b = dpct::blas_gemm::experimental::pointer_mode_t::host;
  // CHECK-NEXT: b = dpct::blas_gemm::experimental::pointer_mode_t::device;
  // CHECK-NEXT: b = dpct::blas_gemm::experimental::pointer_mode_t::device_vector;
  // CHECK-NEXT: b = dpct::blas_gemm::experimental::pointer_mode_t::alpha_device_vector_beta_zero;
  // CHECK-NEXT: b = dpct::blas_gemm::experimental::pointer_mode_t::alpha_device_vector_beta_host;
  hipblasLtPointerMode_t b;
  b = HIPBLASLT_POINTER_MODE_HOST;
  b = HIPBLASLT_POINTER_MODE_DEVICE;
  b = CUBLASLT_POINTER_MODE_DEVICE_VECTOR;
  b = CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;
  b = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
  // CHECK: dpct::blas_gemm::experimental::matrix_layout_t::attribute c;
  // CHECK-NEXT: c = dpct::blas_gemm::experimental::matrix_layout_t::attribute::type;
  // CHECK-NEXT: c = dpct::blas_gemm::experimental::matrix_layout_t::attribute::order;
  // CHECK-NEXT: c = dpct::blas_gemm::experimental::matrix_layout_t::attribute::rows;
  // CHECK-NEXT: c = dpct::blas_gemm::experimental::matrix_layout_t::attribute::cols;
  // CHECK-NEXT: c = dpct::blas_gemm::experimental::matrix_layout_t::attribute::ld;
  hipblasLtMatrixLayoutAttribute_t c;
  c = HIPBLASLT_MATRIX_LAYOUT_TYPE;
  c = HIPBLASLT_MATRIX_LAYOUT_ORDER;
  c = HIPBLASLT_MATRIX_LAYOUT_ROWS;
  c = HIPBLASLT_MATRIX_LAYOUT_COLS;
  c = HIPBLASLT_MATRIX_LAYOUT_LD;
  // CHECK: dpct::blas_gemm::experimental::matmul_desc_t::attribute d;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::compute_type;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::scale_type;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::pointer_mode;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::trans_a;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::trans_b;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::trans_c;
  // CHECK-NEXT: d = dpct::blas_gemm::experimental::matmul_desc_t::attribute::epilogue;
  hipblasLtMatmulDescAttributes_t d;
  d = CUBLASLT_MATMUL_DESC_COMPUTE_TYPE;
  d = CUBLASLT_MATMUL_DESC_SCALE_TYPE;
  d = HIPBLASLT_MATMUL_DESC_POINTER_MODE;
  d = HIPBLASLT_MATMUL_DESC_TRANSA;
  d = HIPBLASLT_MATMUL_DESC_TRANSB;
  d = CUBLASLT_MATMUL_DESC_TRANSC;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE;
  // CHECK: int e;
  // CHECK-NEXT: e = 1;
  // CHECK-NEXT: e = 2;
  // CHECK-NEXT: e = 130;
  // CHECK-NEXT: e = 4;
  // CHECK-NEXT: e = 6;
  // CHECK-NEXT: e = 134;
  // CHECK-NEXT: e = 136;
  // CHECK-NEXT: e = 152;
  // CHECK-NEXT: e = 32;
  // CHECK-NEXT: e = 160;
  // CHECK-NEXT: e = 36;
  // CHECK-NEXT: e = 164;
  // CHECK-NEXT: e = 192;
  // CHECK-NEXT: e = 208;
  // CHECK-NEXT: e = 256;
  // CHECK-NEXT: e = 512;
  hipblasLtEpilogue_t e;
  e = HIPBLASLT_EPILOGUE_DEFAULT;
  e = HIPBLASLT_EPILOGUE_RELU;
  e = HIPBLASLT_EPILOGUE_RELU_AUX;
  e = HIPBLASLT_EPILOGUE_BIAS;
  e = HIPBLASLT_EPILOGUE_RELU_BIAS;
  e = HIPBLASLT_EPILOGUE_RELU_AUX_BIAS;
  e = CUBLASLT_EPILOGUE_DRELU;
  e = CUBLASLT_EPILOGUE_DRELU_BGRAD;
  e = HIPBLASLT_EPILOGUE_GELU;
  e = HIPBLASLT_EPILOGUE_GELU_AUX;
  e = HIPBLASLT_EPILOGUE_GELU_BIAS;
  e = HIPBLASLT_EPILOGUE_GELU_AUX_BIAS;
  e = HIPBLASLT_EPILOGUE_DGELU;
  e = HIPBLASLT_EPILOGUE_DGELU_BGRAD;
  e = HIPBLASLT_EPILOGUE_BGRADA;
  e = HIPBLASLT_EPILOGUE_BGRADB;
}

void foo4() {
  // CHECK: dpct::blas_gemm::experimental::transform_desc_ptr transformDesc;
  // CHECK-NEXT: dpct::library_data_t scaleType;
  // CHECK-NEXT: transformDesc = new dpct::blas_gemm::experimental::transform_desc_t(scaleType);
  // CHECK-NEXT: oneapi::mkl::transpose opT = oneapi::mkl::transpose::trans;
  // CHECK-NEXT: size_t sizeWritten;
  // CHECK-NEXT: transformDesc->set_attribute(dpct::blas_gemm::experimental::transform_desc_t::attribute::trans_a, &opT);
  // CHECK-NEXT: transformDesc->get_attribute(dpct::blas_gemm::experimental::transform_desc_t::attribute::trans_a, &opT);
  // CHECK-NEXT: delete (transformDesc);
  hipblasLtMatrixTransformDesc_t transformDesc;
  hipDataType scaleType;
  hipblasLtMatrixTransformDescCreate(&transformDesc, scaleType);
  hipblasOperation_t opT = HIPBLAS_OP_T;
  size_t sizeWritten;
  hipblasLtMatrixTransformDescSetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opT, sizeof(opT));
  hipblasLtMatrixTransformDescGetAttribute(transformDesc, HIPBLASLT_MATRIX_TRANSFORM_DESC_TRANSA, &opT, sizeof(opT), &sizeWritten);
  hipblasLtMatrixTransformDescDestroy(transformDesc);

  // CHECK: dpct::blas_gemm::experimental::descriptor_ptr lightHandle;
  // CHECK-NEXT: const void *alpha;
  // CHECK-NEXT: const void *A;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Adesc;
  // CHECK-NEXT: const void *beta;
  // CHECK-NEXT: const void *B;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Bdesc;
  // CHECK-NEXT: void *C;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_layout_ptr Cdesc;
  // CHECK-NEXT: dpct::queue_ptr stream;
  // CHECK-NEXT: dpct::blas_gemm::experimental::matrix_transform(transformDesc, alpha, A, Adesc, beta, B, Bdesc, C, Cdesc, stream);
  hipblasLtHandle_t lightHandle;
  const void *alpha;
  const void *A;
  hipblasLtMatrixLayout_t Adesc;
  const void *beta;
  const void *B;
  hipblasLtMatrixLayout_t Bdesc;
  void *C;
  hipblasLtMatrixLayout_t Cdesc;
  hipStream_t stream;
  hipblasLtMatrixTransform(lightHandle, transformDesc, alpha, A, Adesc, beta, B, Bdesc, C, Cdesc, stream);
}
