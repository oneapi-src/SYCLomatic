// RUN: dpct --format-range=none --out-root %T/cublaslt %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/cublaslt/cublaslt.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/cublaslt/cublaslt.dp.cpp -o %T/cublaslt/cublaslt.dp.o %}

#include "hipblaslt.h"

void foo1 () {
  hipblasLtHandle_t ltHandle;
  hipblasLtCreate(&ltHandle);
  hipblasLtDestroy(ltHandle);

  hipblasLtMatrixLayout_t matLayout;
  hipDataType type;
  uint64_t rows;
  uint64_t cols;
  int64_t ld;
  hipblasLtMatrixLayoutCreate(&matLayout, type, rows, cols, ld);

  hipblasLtMatrixLayoutAttribute_t attr1;
  void *buf1;
  size_t sizeInBytes1;
  size_t *sizeWritten1;
  hipblasLtMatrixLayoutGetAttribute(matLayout, attr1, buf1, sizeInBytes1, sizeWritten1);
  hipblasLtMatrixLayoutSetAttribute(matLayout, attr1, buf1, sizeInBytes1);
  hipblasLtMatrixLayoutDestroy(matLayout);

  hipblasLtMatmulDesc_t matmulDesc;
  hipblasComputeType_t computeType;
  hipDataType scaleType;
  hipblasLtMatmulDescCreate(&matmulDesc, computeType, scaleType);

  hipblasLtMatmulDescAttributes_t attr2;
  void *buf2;
  size_t sizeInBytes2;
  size_t *sizeWritten2;
  hipblasLtMatmulDescGetAttribute(matmulDesc, attr2, buf2, sizeInBytes2, sizeWritten2);
  hipblasLtMatmulDescSetAttribute(matmulDesc, attr2, buf2, sizeInBytes2);
  hipblasLtMatmulDescDestroy(matmulDesc);
}

void foo2() {
  hipblasLtHandle_t lightHandle;
  hipblasLtMatmulDesc_t computeDesc;
  const void *alpha;
  const void *A;
  hipblasLtMatrixLayout_t Adesc;
  const void *B;
  hipblasLtMatrixLayout_t Bdesc;
  const void *beta;
  const void *C;
  hipblasLtMatrixLayout_t Cdesc;
  void *D;
  hipblasLtMatrixLayout_t Ddesc;
  const hipblasLtMatmulAlgo_t *algo;
  void *workspace;
  size_t workspaceSizeInBytes;
  hipStream_t stream;
  hipblasLtMatmul(lightHandle, computeDesc, alpha, A, Adesc, B, Bdesc, beta, C, Cdesc, D, Ddesc, algo, workspace, workspaceSizeInBytes, stream);
}

void foo3() {
  hipblasLtOrder_t a;
  a = HIPBLASLT_ORDER_COL;
  a = HIPBLASLT_ORDER_ROW;
  a = CUBLASLT_ORDER_COL32;
  a = CUBLASLT_ORDER_COL4_4R2_8C;
  a = CUBLASLT_ORDER_COL32_2R_4R4;
  hipblasLtPointerMode_t b;
  b = HIPBLASLT_POINTER_MODE_HOST;
  b = HIPBLASLT_POINTER_MODE_DEVICE;
  b = CUBLASLT_POINTER_MODE_DEVICE_VECTOR;
  b = CUBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_ZERO;
  b = HIPBLASLT_POINTER_MODE_ALPHA_DEVICE_VECTOR_BETA_HOST;
  hipblasLtMatrixLayoutAttribute_t c;
  c = HIPBLASLT_MATRIX_LAYOUT_TYPE;
  c = HIPBLASLT_MATRIX_LAYOUT_ORDER;
  c = HIPBLASLT_MATRIX_LAYOUT_ROWS;
  c = HIPBLASLT_MATRIX_LAYOUT_COLS;
  c = HIPBLASLT_MATRIX_LAYOUT_LD;
  c = HIPBLASLT_MATRIX_LAYOUT_BATCH_COUNT;
  c = HIPBLASLT_MATRIX_LAYOUT_STRIDED_BATCH_OFFSET;
  c = CUBLASLT_MATRIX_LAYOUT_PLANE_OFFSET;
  hipblasLtMatmulDescAttributes_t d;
  d = CUBLASLT_MATMUL_DESC_COMPUTE_TYPE;
  d = CUBLASLT_MATMUL_DESC_SCALE_TYPE;
  d = HIPBLASLT_MATMUL_DESC_POINTER_MODE;
  d = HIPBLASLT_MATMUL_DESC_TRANSA;
  d = HIPBLASLT_MATMUL_DESC_TRANSB;
  d = CUBLASLT_MATMUL_DESC_TRANSC;
  d = CUBLASLT_MATMUL_DESC_FILL_MODE;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE;
  d = HIPBLASLT_MATMUL_DESC_BIAS_POINTER;
  d = CUBLASLT_MATMUL_DESC_BIAS_BATCH_STRIDE;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_POINTER;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_LD;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_BATCH_STRIDE;
  d = CUBLASLT_MATMUL_DESC_ALPHA_VECTOR_BATCH_STRIDE;
  d = CUBLASLT_MATMUL_DESC_SM_COUNT_TARGET;
  d = HIPBLASLT_MATMUL_DESC_A_SCALE_POINTER;
  d = HIPBLASLT_MATMUL_DESC_B_SCALE_POINTER;
  d = HIPBLASLT_MATMUL_DESC_C_SCALE_POINTER;
  d = HIPBLASLT_MATMUL_DESC_D_SCALE_POINTER;
  d = HIPBLASLT_MATMUL_DESC_AMAX_D_POINTER;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_DATA_TYPE;
  d = HIPBLASLT_MATMUL_DESC_EPILOGUE_AUX_SCALE_POINTER;
  d = CUBLASLT_MATMUL_DESC_EPILOGUE_AUX_AMAX_POINTER;
  d = CUBLASLT_MATMUL_DESC_FAST_ACCUM;
  d = HIPBLASLT_MATMUL_DESC_BIAS_DATA_TYPE;
  d = CUBLASLT_MATMUL_DESC_ATOMIC_SYNC_IN_COUNTERS_POINTER;
  d = CUBLASLT_MATMUL_DESC_ATOMIC_SYNC_OUT_COUNTERS_POINTER;
  d = CUBLASLT_MATMUL_DESC_ATOMIC_SYNC_NUM_CHUNKS_D_ROWS;
  d = CUBLASLT_MATMUL_DESC_ATOMIC_SYNC_NUM_CHUNKS_D_COLS;
}
