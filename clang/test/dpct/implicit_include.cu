
#include <hip/hip_runtime.h>
// RUN: dpct -out-root %T/implicit_include %s --cuda-include-path="%cuda-path/include"
// RUN: FileCheck --input-file %T/implicit_include/implicit_include.dp.cpp --match-full-lines %s
// RUN: %if build_lit %{icpx -c -fsycl %T/implicit_include/implicit_include.dp.cpp -o %T/implicit_include/implicit_include.dp.o %}

// CHECK: #include <cmath>

void test() {
  int a = -1;
  // CHECK: abs(a);
  abs(a);
}
