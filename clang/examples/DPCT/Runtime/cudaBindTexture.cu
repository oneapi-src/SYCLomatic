// Option: --use-experimental-features=bindless_images

void test(size_t *ps, const textureReference *ptr, const void *pv,
          const hipChannelFormatDesc *pc, size_t s) {
  // Start
  hipBindTexture(ps /*size_t **/, ptr /*const textureReference **/,
                  pv /*const void **/, pc /*const hipChannelFormatDesc **/,
                  s /*size_t*/);
  // End
}
