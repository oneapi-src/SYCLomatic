
#include <hip/hip_runtime.h>
void test(void *pv, size_t s1, const void *cpv, size_t s2, size_t s3, size_t s4,
          hipMemcpyKind m, hipStream_t s) {
  // Start
  hipMemcpy2DAsync(pv /*void **/, s1 /*size_t*/, cpv /*const void **/,
                    s2 /*size_t*/, s3 /*size_t*/, s4 /*size_t*/,
                    m /*cudaMemoryAdvise*/, s /*cudaStream_t*/);
  // End
}
