
#include <hip/hip_runtime.h>
void test(void *pv, const void *cpv, size_t s, hipMemcpyKind m) {
  // Start
  hipMemcpy(pv /*void **/, cpv /*const void **/, s /*size_t*/,
             m /*cudaMemcpyKind*/);
  // End
}
