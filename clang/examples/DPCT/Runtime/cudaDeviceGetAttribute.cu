
#include <hip/hip_runtime.h>
void test(int *pi, int i) {
  // Start
  // Only support migration of some cudaDeviceAttr type.
  /* 1 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeMaxThreadsPerBlock
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 2 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeClockRate
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 3 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeTextureAlignment
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 4 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeMultiprocessorCount
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 5 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeIntegrated
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 6 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeComputeMode
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 7 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeComputeCapabilityMajor
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 8 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeComputeCapabilityMinor
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 9 */ hipDeviceGetAttribute(pi /*int **/,
                                 hipDeviceAttributeHostNativeAtomicSupported
                                 /*cudaDeviceAttr*/,
                                 i /*int*/);
  /* 10 */ hipDeviceGetAttribute(pi /*int **/,
                                  hipDeviceAttributeConcurrentManagedAccess
                                  /*cudaDeviceAttr*/,
                                  i /*int*/);
  // End
}
