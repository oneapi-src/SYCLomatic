
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

void test(int c, hipGraphicsResource_t *r, hipStream_t s) {
  // Start
  hipGraphicsUnmapResources(c /*int*/,
                             r /*cudaGraphicsResource_t **/);
  hipGraphicsUnmapResources(c /*int*/,
                             r /*cudaGraphicsResource_t **/,
                             s /*cudaStream_t*/);
  // End
}
