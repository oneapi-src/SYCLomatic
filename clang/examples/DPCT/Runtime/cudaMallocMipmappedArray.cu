
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

void test(hipMipmappedArray_t *m, const hipChannelFormatDesc *d, hipExtent e,
          unsigned u1, unsigned u2) {
  // Start
  hipMallocMipmappedArray(m /*cudaMipmappedArray_t **/,
                           d /*const cudaChannelFormatDesc **/,
                           e /*cudaExtent*/, u1 /*unsigned*/, u2 /*unsigned*/);
  // End
}
