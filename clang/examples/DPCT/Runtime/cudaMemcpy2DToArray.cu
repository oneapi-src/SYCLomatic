
#include <hip/hip_runtime.h>
void test(hipArray_t a, size_t s1, size_t s2, const void *pv, size_t s3,
          size_t s4, size_t s5, hipMemcpyKind m) {
  // Start
  hipMemcpy2DToArray(a /*cudaArray_t*/, s1 /*size_t*/, s2 /*size_t*/,
                      pv /*const void **/, s3 /*size_t*/, s4 /*size_t*/,
                      s5 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
