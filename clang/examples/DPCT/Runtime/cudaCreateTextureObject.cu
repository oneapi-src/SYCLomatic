
#include <hip/hip_runtime.h>
void test(hipTextureObject_t *pto, const hipResourceDesc *prd,
          const hipTextureDesc *ptd, const hipResourceViewDesc *prvd) {
  // Start
  hipCreateTextureObject(
      pto /*cudaTextureObject_t **/, prd /*const cudaResourceDesc **/,
      ptd /*const cudaTextureDesc **/, prvd /*const cudaResourceViewDesc **/);
  // End
}
