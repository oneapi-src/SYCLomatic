
#include <hip/hip_runtime.h>
void test(const void *cpv1, const void *cpv2, size_t s1, size_t s2,
          hipMemcpyKind m) {
  // Start
  hipStream_t s;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(cpv1) /*const void **/, cpv2 /*const void **/,
                          s1 /*size_t*/, s2 /*size_t*/, m /*cudaMemcpyKind*/,
                          s /*cudaStream_t*/);
  // End
}
