
#include <hip/hip_runtime.h>
void test(const void *symbol, const void *src, size_t s1, size_t s2,
          hipMemcpyKind m) {
  // Start
  hipStream_t s;
  hipMemcpyToSymbolAsync(HIP_SYMBOL(symbol) /*const void **/, src /*const void **/,
                          s1 /*size_t*/, s2 /*size_t*/, m /*cudaMemcpyKind*/,
                          s);
  // End
}
