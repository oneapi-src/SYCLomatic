
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=occupancy-calculation
void test(int *pi, const void *pv, int i, size_t s) {
  // Start
  hipOccupancyMaxActiveBlocksPerMultiprocessor(
      pi /*int **/, pv /*const void **/, i /*int*/, s /*size_t*/);
  // End
}
