
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=masked-sub-group-operation
__device__ void test(unsigned int r, unsigned int mask, unsigned int var,
                     int src_lane, int width) {
  // Start
  r = __shfl_sync(mask /*unsigned int*/, var /*unsigned int*/, src_lane /*int*/,
                  width /*int*/);
  // End
}