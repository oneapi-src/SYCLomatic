
#include <hip/hip_runtime.h>
void test(void *pv, const void *cpv, size_t s1, size_t s2, hipMemcpyKind m) {
  // Start
  hipMemcpyFromSymbol(pv /*void **/, HIP_SYMBOL(cpv) /*const void **/, s1 /*size_t*/,
                       s2 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
