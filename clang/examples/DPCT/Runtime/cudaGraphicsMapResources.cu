
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

void test(int c, hipGraphicsResource_t *r, hipStream_t s) {
  // Start
  hipGraphicsMapResources(c /*int*/,
                           r /*cudaGraphicsResource_t **/);
  hipGraphicsMapResources(c /*int*/,
                           r /*cudaGraphicsResource_t **/,
                           s /*cudaStream_t*/);
  // End
}
