
#include <hip/hip_runtime.h>
void test(const void *f, dim3 gridDim, dim3 blockDim, void **args,
          size_t sharedMem, hipStream_t s) {
  // Start
  hipLaunchKernel(reinterpret_cast<const void*>(f) /*cudaError_t*/, gridDim /*dim3*/, blockDim /*dim3*/,
                   args /*void ***/, sharedMem /*size_t*/, s /*cudaStream_t*/);
  // End
}
