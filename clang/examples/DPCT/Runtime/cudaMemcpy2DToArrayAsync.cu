
#include <hip/hip_runtime.h>
void test(size_t s1, size_t s2, const void *src, size_t s3, size_t s4,
          size_t s5, hipMemcpyKind m) {
  // Start
  hipArray_t dst;
  hipStream_t s;
  hipMemcpy2DToArrayAsync(dst, s1 /*size_t*/, s2 /*size_t*/,
                           src /*const void **/, s3 /*size_t*/, s4 /*size_t*/,
                           s5 /*size_t*/, m /*cudaMemcpyKind*/, s);
  // End
}
