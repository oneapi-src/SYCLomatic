
#include <hip/hip_runtime.h>
void test(hipArray_t a, size_t s1, size_t s2, const void *pv, size_t s3,
          size_t s4, size_t s5, hipMemcpyKind m, hipStream_t s) {
  // Start
  hipMemcpy2DToArrayAsync(a /*cudaArray_t*/, s1 /*size_t*/, s2 /*size_t*/,
                           pv /*const void **/, s3 /*size_t*/, s4 /*size_t*/,
                           s5 /*size_t*/, m /*cudaMemcpyKind*/,
                           s /*cudaStream_t*/);
  // End
}
