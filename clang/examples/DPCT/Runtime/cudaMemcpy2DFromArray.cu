
#include <hip/hip_runtime.h>
void test(void *pv, size_t s1, hipArray_const_t a, size_t s2, size_t s3,
          size_t s4, size_t s5, hipMemcpyKind m) {
  // Start
  hipMemcpy2DFromArray(pv /*void **/, s1 /*size_t*/, a /*cudaArray_const_t*/,
                        s2 /*size_t*/, s3 /*size_t*/, s4 /*size_t*/,
                        s5 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
