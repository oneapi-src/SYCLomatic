#include "hip/hip_runtime.h"
#include "hip/hip_cooperative_groups.h"
#include <cooperative_groups/reduce.h>


__global__ void test(int *sdata, const cooperative_groups::thread_block &cta) {
    // Start
    const unsigned int tid = cta.thread_rank();
    cooperative_groups::thread_block_tile<32> tile32 = cooperative_groups::tiled_partition<32>(cta);
    cooperative_groups::reduce(tile32 /* cooperative_groups::thread_block_til */, sdata[tid] /* void */, cooperative_groups::less<int>() /* */);
    // End
}
