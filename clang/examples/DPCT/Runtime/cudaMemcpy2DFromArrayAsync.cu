
#include <hip/hip_runtime.h>
void test(void *pv, size_t s1, hipArray_const_t a, size_t s2, size_t s3,
          size_t s4, size_t s5, hipMemcpyKind m, hipStream_t s) {
  // Start
  hipMemcpy2DFromArrayAsync(pv /*void **/, s1 /*size_t*/,
                             a /*cudaArray_const_t*/, s2 /*size_t*/,
                             s3 /*size_t*/, s4 /*size_t*/, s5 /*size_t*/,
                             m /*cudaMemcpyKind*/, s /*cudaStream_t*/);
  // End
}
