
#include <hip/hip_runtime.h>
void test(void *dst, size_t s1, size_t s2, size_t s3, size_t s4, size_t s5,
          hipMemcpyKind m) {
  // TODO: src's type need to be changed to cudaArray_const_t
  // Start
  hipArray_t src;
  hipStream_t s;
  hipMemcpy2DFromArrayAsync(dst /*void **/, s1 /*size_t*/, src, s2 /*size_t*/,
                             s3 /*size_t*/, s4 /*size_t*/, s5 /*size_t*/,
                             m /*cudaMemcpyKind*/, s);
  // End
}
