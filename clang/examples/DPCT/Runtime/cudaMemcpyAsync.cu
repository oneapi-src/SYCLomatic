
#include <hip/hip_runtime.h>
void test(void *pv, const void *cpv, size_t s, hipMemcpyKind m) {
  // Start
  hipStream_t cs;
  hipMemcpyAsync(pv /*void **/, cpv /*const void **/, s /*size_t*/,
                  m /*cudaMemcpyKind*/, cs /*cudaStream_t*/);
  // End
}
