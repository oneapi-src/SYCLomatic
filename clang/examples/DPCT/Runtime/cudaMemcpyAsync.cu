
#include <hip/hip_runtime.h>
void test(void *dst, const void *src, size_t s, hipMemcpyKind m) {
  // Start
  hipStream_t cs;
  hipMemcpyAsync(dst /*void **/, src /*const void **/, s /*size_t*/,
                  m /*cudaMemcpyKind*/, cs);
  // End
}
