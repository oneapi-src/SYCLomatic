
#include <hip/hip_runtime.h>
void test(void *pv, const void *cpv, size_t s1, size_t s2, hipMemcpyKind m) {
  // Start
  hipStream_t s;
  hipMemcpyFromSymbolAsync(pv /*void **/, HIP_SYMBOL(cpv) /*const void **/, s1 /*size_t*/,
                            s2 /*size_t*/, m /*cudaMemcpyKind*/,
                            s /*cudaStream_t*/);
  // End
}
