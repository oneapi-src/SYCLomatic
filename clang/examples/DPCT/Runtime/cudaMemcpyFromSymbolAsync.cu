
#include <hip/hip_runtime.h>
void test(void *dst, const void *symbol, size_t s1, size_t s2,
          hipMemcpyKind m) {
  // Start
  hipStream_t s;
  hipMemcpyFromSymbolAsync(dst /*void **/, HIP_SYMBOL(symbol) /*const void **/,
                            s1 /*size_t*/, s2 /*size_t*/, m /*cudaMemcpyKind*/,
                            s);
  // End
}
