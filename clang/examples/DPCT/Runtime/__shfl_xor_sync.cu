
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=masked-sub-group-operation
__device__ void test(unsigned int r, unsigned int mask, unsigned int var,
                     int lane, int width) {
  // Start
  r = __shfl_xor_sync(mask /*unsigned int*/, var /*unsigned int*/, lane /*int*/,
                      width /*int*/);
  // End
}