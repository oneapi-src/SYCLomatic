
#include <hip/hip_runtime.h>
void test(hipArray_t a, size_t s1, size_t s2, const void *pv, size_t s3,
          hipMemcpyKind m, hipStream_t s) {
  // Start
  cudaMemcpyToArrayAsync(a /*cudaArray_t*/, s1 /*size_t*/, s2 /*size_t*/,
                         pv /*const void **/, s3 /*size_t*/,
                         m /*cudaMemcpyKind*/, s /*cudaStream_t*/);
  // End
}
