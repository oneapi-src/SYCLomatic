
#include <hip/hip_runtime.h>
void test(size_t s1, size_t s2, const void *src, size_t s3, hipMemcpyKind m) {
  // Start
  hipArray_t dst;
  hipStream_t s;
  cudaMemcpyToArrayAsync(dst, s1 /*size_t*/, s2 /*size_t*/, src /*const void **/,
                         s3 /*size_t*/, m /*cudaMemcpyKind*/, s);
  // End
}
