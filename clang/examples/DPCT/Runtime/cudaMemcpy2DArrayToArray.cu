#define __host__ // May insert these code in dpct binary.
struct size_t {};
struct hipError_t {};
struct hipArray_t {};
struct hipMemcpyKind {};
extern __host__ hipError_t hipMemcpy2DArrayToArray(hipArray_t, size_t,
                                                     size_t, hipArray_t,
                                                     size_t, size_t, size_t,
                                                     size_t, hipMemcpyKind);

void test(size_t s1, size_t s2, size_t s3, size_t s4, size_t s5, size_t s6) {
  // TODO: src's type need to be changed to hipArray_const_t
  // Start
  hipArray_t dst;
  hipArray_t src;
  hipMemcpyKind m;
  hipMemcpy2DArrayToArray(dst, s1 /*size_t*/, s2 /*size_t*/, src,
                           s3 /*size_t*/, s4 /*size_t*/, s5 /*size_t*/,
                           s6 /*size_t*/, m);
  // End
}
