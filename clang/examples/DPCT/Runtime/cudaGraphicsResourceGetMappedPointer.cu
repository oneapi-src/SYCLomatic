
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

void test(void *ptr, size_t *s, hipGraphicsResource_t r) {
  // Start
  hipGraphicsResourceGetMappedPointer(&ptr /*void ***/,
                                        s /*size_t **/,
                                        r /*cudaGraphicsResource_t*/);
  // End
}
