// Option: --use-experimental-features=bindless_images

void test(const textureReference *ptr, const hipArray_t a,
          const hipChannelFormatDesc *pc) {
  // Start
  hipBindTextureToArray(ptr /*const textureReference **/,
                         a /*const hipArray_t*/,
                         pc /*const hipChannelFormatDesc **/);
  // End
}
