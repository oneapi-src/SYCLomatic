
#include <hip/hip_runtime.h>
void test(void *pv, hipArray_const_t a, size_t s1, size_t s2, size_t s3,
          hipMemcpyKind m) {
  // Start
  hipMemcpyFromArray(pv /*void **/, a /*cudaArray_const_t*/, s1 /*size_t*/,
                      s2 /*size_t*/, s3 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
