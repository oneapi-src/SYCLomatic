
#include <hip/hip_runtime.h>
void test(hipArray_t a, size_t s1, size_t s2, const void *pv, size_t s3,
          hipMemcpyKind m) {
  // Start
  hipMemcpyToArray(a /*cudaArray_t*/, s1 /*size_t*/, s2 /*size_t*/,
                    pv /*const void **/, s3 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
