
#include <hip/hip_runtime.h>
void test(const void *cpv1, const void *cpv2, size_t s1, size_t s2,
          hipMemcpyKind m) {
  // Start
  hipMemcpyToSymbol(HIP_SYMBOL(cpv1) /*const void **/, cpv2 /*const void **/,
                     s1 /*size_t*/, s2 /*size_t*/, m /*cudaMemcpyKind*/);
  // End
}
