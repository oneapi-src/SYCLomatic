
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

template <typename T>
__global__ void test(hipTextureObject_t t, float f1, float f2) {
  // Start
  tex1DLod<T>(t /*cudaTextureObject_t*/, f1 /*float*/, f2 /*float*/);
  // End
}
