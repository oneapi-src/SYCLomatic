
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=bindless_images

template <typename T>
__global__ void test(hipTextureObject_t t, float f1, float f2, float f3) {
  // Start
  tex2DLod<T>(t /*cudaTextureObject_t*/, f1 /*float*/, f2 /*float*/,
              f3 /*float*/);
  // End
}
