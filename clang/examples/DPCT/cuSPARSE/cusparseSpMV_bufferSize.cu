
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseDnVecDescr_t vec_x,
          const void *beta, hipsparseDnVecDescr_t vec_y,
          hipDataType compute_type, hipsparseSpMVAlg_t alg) {
  // Start
  size_t buffer_size;
  hipsparseSpMV_bufferSize(
      handle /*cusparseHandle_t*/, trans /*cusparseOperation_t*/,
      alpha /*const void **/, mat_a /*cusparseSpMatDescr_t*/,
      vec_x /*cusparseDnVecDescr_t*/, beta /*const void **/,
      vec_y /*cusparseDnVecDescr_t*/, compute_type /*cudaDataType*/,
      alg /*cusparseSpMVAlg_t*/, &buffer_size /*size_t **/);
  // End
}
