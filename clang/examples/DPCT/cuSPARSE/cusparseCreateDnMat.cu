
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(int64_t rows, int64_t cols, int64_t ld, void *value,
          hipDataType value_type, hipsparseOrder_t order) {
  // Start
  hipsparseDnMatDescr_t desc;
  hipsparseCreateDnMat(&desc /*cusparseDnMatDescr_t **/, rows /*int64_t*/,
                      cols /*int64_t*/, ld /*int64_t*/, value /*void **/,
                      value_type /*cudaDataType*/, order /*cusparseOrder_t*/);
  // End
}
