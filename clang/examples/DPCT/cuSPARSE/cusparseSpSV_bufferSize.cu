
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseDnVecDescr_t vec_x,
          hipsparseDnVecDescr_t vec_y, hipDataType compute_type,
          hipsparseSpSVAlg_t alg, hipsparseSpSVDescr_t desc) {
  // Start
  size_t buffer_size;
  hipsparseSpSV_bufferSize(
      handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
      alpha /*const void **/, mat_a /*cusparseSpMatDescr_t*/,
      vec_x /*cusparseDnVecDescr_t*/, vec_y /*cusparseDnVecDescr_t*/,
      compute_type /*cudaDataType*/, alg /*cusparseSpSVAlg_t*/,
      desc /*cusparseSpSVDescr_t*/, &buffer_size /*size_t **/);
  // End
}
