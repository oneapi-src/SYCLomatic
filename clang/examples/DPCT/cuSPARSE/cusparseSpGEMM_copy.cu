
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a,
          hipsparseOperation_t op_b, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseSpMatDescr_t mat_b,
          const void *beta, hipsparseSpMatDescr_t mat_c,
          hipDataType compute_type, hipsparseSpGEMMAlg_t alg,
          hipsparseSpGEMMDescr_t desc) {
  // Start
  hipsparseSpGEMM_copy(
      handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
      op_b /*cusparseOperation_t*/, alpha /*const void **/,
      mat_a /*cusparseSpMatDescr_t*/, mat_b /*cusparseSpMatDescr_t*/,
      beta /*const void **/, mat_c /*cusparseSpMatDescr_t*/,
      compute_type /*cudaDataType*/, alg /*cusparseSpGEMMAlg_t*/,
      desc /*cusparseSpGEMMDescr_t*/);
  // End
}
