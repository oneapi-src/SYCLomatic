
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a,
          hipsparseOperation_t op_b, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseDnMatDescr_t mat_b,
          hipsparseDnMatDescr_t mat_c, hipDataType compute_type,
          hipsparseSpSMAlg_t alg, hipsparseSpSMDescr_t desc) {
  // Start
  size_t buffer_size;
  hipsparseSpSM_bufferSize(
      handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
      op_b /*cusparseOperation_t*/, alpha /*const void **/,
      mat_a /*cusparseSpMatDescr_t*/, mat_b /*cusparseSpMatDescr_t*/,
      mat_c /*cusparseSpMatDescr_t*/, compute_type /*cudaDataType*/,
      alg /*cusparseSpSMAlg_t*/, desc /*cusparseSpSMDescr_t*/,
      &buffer_size /*size_t **/);
  // End
}
