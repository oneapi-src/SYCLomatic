
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseSpMatDescr_t desc, void *row_ptr, void *col_ind,
          void *value) {
  // Start
  hipsparseCsrSetPointers(desc /*cusparseSpMatDescr_t*/, row_ptr /*void **/,
                         col_ind /*void **/, value /*void **/);
  // End
}
