
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(int64_t size, void *value, hipDataType value_type) {
  // Start
  hipsparseConstDnVecDescr_t desc;
  hipsparseCreateConstDnVec(&desc /*cusparseConstDnVecDescr_t **/,
                           size /*int64_t*/, value /*void **/,
                           value_type /*cudaDataType*/);
  // End
}
