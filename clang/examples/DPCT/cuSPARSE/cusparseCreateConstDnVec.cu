
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(int64_t size, const void *value, hipDataType value_type) {
  // Start
  hipsparseConstDnVecDescr_t desc;
  hipsparseCreateConstDnVec(&desc /*cusparseConstDnVecDescr_t **/,
                           size /*int64_t*/, value /*const void **/,
                           value_type /*cudaDataType*/);
  // End
}
