#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m,
          const hipComplex *alpha, hipsparseMatDescr_t desc,
          const hipComplex *value, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, const hipComplex *f, hipComplex *x) {
  // Start
  cusparseCcsrsv_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      alpha /*const hipComplex **/, desc /*hipsparseMatDescr_t*/,
      value /*const hipComplex **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/,
      f /*const hipComplex **/, x /*hipComplex **/);
  // End
}
