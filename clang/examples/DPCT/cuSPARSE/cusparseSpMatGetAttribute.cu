
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseSpMatDescr_t desc, hipsparseSpMatAttribute_t attr, void *data,
          size_t data_size) {
  // Start
  hipsparseSpMatGetAttribute(desc /*cusparseSpMatDescr_t*/,
                            attr /*cusparseSpMatAttribute_t*/, data /*void **/,
                            data_size /*size_t*/);
  // End
}
