#include "hipsparse.h"

void test(hipsparseHandle_t handle, int m, int n, int nnz,
          const float *csr_value, const int *row_ptr, const int *col_idx,
          float *csc_value, int *row_ind, int *col_ptr, hipsparseAction_t act,
          hipsparseIndexBase_t base) {
  // Start
  hipsparseScsr2csc(handle /*hipsparseHandle_t*/, m /*int*/, n /*int*/,
                   nnz /*int*/, csr_value /*const float **/,
                   row_ptr /*const int **/, col_idx /*const int **/,
                   csc_value /*float **/, row_ind /*int **/, col_ptr /*int **/,
                   act /*hipsparseAction_t*/, base /*hipsparseIndexBase_t*/);
  // End
}
