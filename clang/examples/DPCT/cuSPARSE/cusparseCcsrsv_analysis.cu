#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const void *alpha, const hipsparseMatDescr_t desc,
          const hipComplex *value, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info) {
  // Start
  cusparseCcsrsv_analysis(handle, trans, m, nnz, desc, value, row_ptr, col_idx,
                          info);
  // End
}
