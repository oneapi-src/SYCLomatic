#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, const hipComplex *value, const int *row_ptr,
          const int *col_idx, cusparseSolveAnalysisInfo_t info) {
  // Start
  cusparseCcsrsv_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*const hipComplex **/,
      row_ptr /*const int **/, col_idx /*const int **/,
      info /*cusparseSolveAnalysisInfo_t*/);
  // End
}
