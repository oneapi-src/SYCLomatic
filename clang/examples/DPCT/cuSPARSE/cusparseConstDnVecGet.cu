#include "hipsparse.h"

void test(hipsparseConstDnVecDescr_t desc) {
  // Start
  int64_t size;
  void *value;
  hipDataType value_type;
  hipsparseConstDnVecGet(desc /*hipsparseConstDnVecDescr_t*/, &size /*int64_t **/,
                        &value /*void ***/, &value_type /*hipDataType **/);
  // End
}
