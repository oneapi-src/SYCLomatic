
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseConstDnVecDescr_t desc) {
  // Start
  int64_t size;
  const void *value;
  hipDataType value_type;
  hipsparseConstDnVecGet(desc /*cusparseConstDnVecDescr_t*/, &size /*int64_t **/,
                        &value /*const void ***/,
                        &value_type /*cudaDataType **/);
  // End
}