#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans_a,
          hipsparseOperation_t trans_b, int m, int n, int k, int nnz,
          const hipDoubleComplex *alpha, const hipsparseMatDescr_t desc,
          const hipDoubleComplex *value, const int *row_ptr, const int *col_idx,
          const hipDoubleComplex *B, int ldb, const hipDoubleComplex *beta,
          hipDoubleComplex *C, int ldc) {
  // Start
  hipsparseZcsrmm2(handle /*hipsparseHandle_t*/, trans_a /*hipsparseOperation_t*/,
                  trans_b /*hipsparseOperation_t*/, m /*int*/, n /*int*/,
                  k /*int*/, nnz /*int*/, alpha /*const hipDoubleComplex **/,
                  desc /*const hipsparseMatDescr_t*/,
                  value /*const hipDoubleComplex **/, row_ptr /*const int **/,
                  col_idx /*const int **/, B /*const hipDoubleComplex **/,
                  ldb /*int*/, beta /*const hipDoubleComplex **/,
                  C /*hipDoubleComplex **/, ldc /*int*/);
  // End
}
