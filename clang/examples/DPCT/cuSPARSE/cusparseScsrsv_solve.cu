#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m,
          const float *alpha, const hipsparseMatDescr_t desc, const float *value,
          const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, const float *f, float *x) {
  // Start
  cusparseScsrsv_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      alpha /*const float **/, desc /*const hipsparseMatDescr_t*/,
      value /*const float **/, row_ptr /*const int **/, col_idx /*const int **/,
      info /*cusparseSolveAnalysisInfo_t*/, f /*const float **/, x /*float **/);
  // End
}
