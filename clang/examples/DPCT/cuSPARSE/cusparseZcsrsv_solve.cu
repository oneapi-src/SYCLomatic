#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m,
          const hipDoubleComplex *alpha, const hipsparseMatDescr_t desc,
          const hipDoubleComplex *value, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, const hipDoubleComplex *f,
          hipDoubleComplex *x) {
  // Start
  cusparseZcsrsv_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      alpha /*const hipDoubleComplex **/, desc /*const hipsparseMatDescr_t*/,
      value /*const hipDoubleComplex **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/,
      f /*const hipDoubleComplex **/, x /*hipDoubleComplex **/);
  // End
}
