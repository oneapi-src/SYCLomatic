#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const hipsparseMatDescr_t desc, const float *value, const int *row_ptr,
          const int *col_idx, cusparseSolveAnalysisInfo_t info) {
  // Start
  cusparseScsrsv_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*const hipsparseMatDescr_t*/, value /*const float **/,
      row_ptr /*const int **/, col_idx /*const int **/,
      info /*cusparseSolveAnalysisInfo_t*/);
  // End
}
