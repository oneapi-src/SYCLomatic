#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, float *value, const int *row_ptr,
          const int *con_ind, csrsv2Info_t info) {
  // Start
  size_t buffer_size;
  hipsparseScsrsv2_bufferSizeExt(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*float **/,
      row_ptr /*const int **/, con_ind /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size /*size_t **/);
  // End
}
