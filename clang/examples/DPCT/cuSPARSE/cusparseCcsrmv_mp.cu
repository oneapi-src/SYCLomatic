#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int nnz, const hipComplex *alpha, const hipsparseMatDescr_t desc,
          const hipComplex *value, const int *row_ptr, const int *col_idx,
          const hipComplex *x, const hipComplex *beta, hipComplex *y) {
  // Start
  cusparseCcsrmv_mp(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      n /*int*/, nnz /*int*/, alpha /*const hipComplex **/,
      desc /*const hipsparseMatDescr_t*/, value /*const hipComplex **/,
      row_ptr /*const int **/, col_idx /*const int **/, x /*const hipComplex **/,
      beta /*const hipComplex **/, y /*hipComplex **/);
  // End
}
