#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int k, int nnz, const hipComplex *alpha, const hipsparseMatDescr_t desc,
          const hipComplex *value, const int *row_ptr, const int *col_idx,
          const hipComplex *B, int ldb, const hipComplex *beta, hipComplex *C,
          int ldc) {
  // Start
  hipsparseCcsrmm(handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/,
                 m /*int*/, n /*int*/, k /*int*/, nnz /*int*/,
                 alpha /*const hipComplex **/, desc /*const hipsparseMatDescr_t*/,
                 value /*const hipComplex **/, row_ptr /*const int **/,
                 col_idx /*const int **/, B /*const hipComplex **/, ldb /*int*/,
                 beta /*const hipComplex **/, C /*hipComplex **/, ldc /*int*/);
  // End
}
