#include "hipsparse.h"

void test(hipsparseHandle_t handle, cusparseAlgMode_t algo,
          hipsparseOperation_t trans, int m, int n, int nnz, const void *alpha,
          hipDataType alpha_type, const hipsparseMatDescr_t desc,
          const void *value, hipDataType value_type, const int *row_ptr,
          const int *col_idx, const void *x, hipDataType x_type,
          const void *beta, hipDataType beta_type, void *y,
          hipDataType y_type, hipDataType exec_type, void *buffer) {
  // Start
  cusparseCsrmvEx(
      handle /*hipsparseHandle_t*/, algo /*cusparseAlgMode_t*/,
      trans /*hipsparseOperation_t*/, m /*int*/, n /*int*/, nnz /*int*/,
      alpha /*const void **/, alpha_type /*hipDataType*/,
      desc /*const hipsparseMatDescr_t*/, value /*const void **/,
      value_type /*hipDataType*/, row_ptr /*const int **/,
      col_idx /*const int **/, x /*const void **/, x_type /*hipDataType*/,
      beta /*const void **/, beta_type /*hipDataType*/, y /*void **/,
      y_type /*hipDataType*/, exec_type /*hipDataType*/, buffer /*void **/);
  // End
}
