
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a,
          hipsparseOperation_t op_b, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseDnMatDescr_t mat_b,
          hipsparseDnMatDescr_t mat_c, hipDataType compute_type,
          hipsparseSpSMAlg_t alg, hipsparseSpSMDescr_t desc, void *buffer) {
  // Start
  hipsparseSpSM_analysis(
      handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
      op_b /*cusparseOperation_t*/, alpha /*const void **/,
      mat_a /*cusparseSpMatDescr_t*/, mat_b /*cusparseSpMatDescr_t*/,
      mat_c /*cusparseSpMatDescr_t*/, compute_type /*cudaDataType*/,
      alg /*cusparseSpSMAlg_t*/, desc /*cusparseSpSMDescr_t*/,
      buffer /*void **/);
  // End
}
