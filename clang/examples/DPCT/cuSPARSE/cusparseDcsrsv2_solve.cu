#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const double *alpha, hipsparseMatDescr_t desc, const double *value,
          const int *row_ptr, const int *col_idx, csrsv2Info_t info,
          const double *f, double *x, hipsparseSolvePolicy_t policy,
          void *buffer) {
  // Start
  hipsparseDcsrsv2_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, alpha /*const double **/, desc /*hipsparseMatDescr_t*/,
      value /*const double **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*csrsv2Info_t*/, f /*const double **/,
      x /*double **/, policy /*hipsparseSolvePolicy_t*/, buffer /*void **/);
  // End
}
