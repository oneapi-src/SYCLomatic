#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int k, int nnz, const hipDoubleComplex *alpha,
          const hipsparseMatDescr_t desc, const hipDoubleComplex *value,
          const int *row_ptr, const int *col_idx, const hipDoubleComplex *B,
          int ldb, const hipDoubleComplex *beta, hipDoubleComplex *C, int ldc) {
  // Start
  hipsparseZcsrmm(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      n /*int*/, k /*int*/, nnz /*int*/, alpha /*const hipDoubleComplex **/,
      desc /*const hipsparseMatDescr_t*/, value /*const hipDoubleComplex **/,
      row_ptr /*const int **/, col_idx /*const int **/,
      B /*const hipDoubleComplex **/, ldb /*int*/,
      beta /*const hipDoubleComplex **/, C /*hipDoubleComplex **/, ldc /*int*/);
  // End
}
