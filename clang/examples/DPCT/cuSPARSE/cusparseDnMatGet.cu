
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseDnMatDescr_t desc) {
  // Start
  int64_t rows;
  int64_t cols;
  int64_t ld;
  void *value;
  hipDataType value_type;
  hipsparseOrder_t order;
  hipsparseDnMatGet(desc /*cusparseDnMatDescr_t*/, &rows /*int64_t **/,
                   &cols /*int64_t **/, &ld /*int64_t **/, &value /*void ***/,
                   &value_type /*cudaDataType **/,
                   &order /*cusparseOrder_t **/);
  // End
}
