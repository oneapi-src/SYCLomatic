#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const float *alpha, const hipsparseMatDescr_t desc, const float *value,
          const int *row_ptr, const int *col_idx, csrsv2Info_t info,
          const float *f, float *x, hipsparseSolvePolicy_t policy,
          void *buffer) {
  // Start
  hipsparseScsrsv2_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, alpha /*const float **/, desc /*const hipsparseMatDescr_t*/,
      value /*const float **/, row_ptr /*const int **/, col_idx /*const int **/,
      info /*csrsv2Info_t*/, f /*const float **/, x /*float **/,
      policy /*hipsparseSolvePolicy_t*/, buffer /*void **/);
  // End
}
