#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, hipComplex *value, const int *row_ptr,
          const int *con_ind, csrsv2Info_t info) {
  // Start
  size_t buffer_size;
  hipsparseCcsrsv2_bufferSizeExt(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*hipComplex **/,
      row_ptr /*const int **/, con_ind /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size /*size_t **/);
  // End
}
