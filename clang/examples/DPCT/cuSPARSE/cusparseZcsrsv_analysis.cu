#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, const hipDoubleComplex *value,
          const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info) {
  // Start
  cusparseZcsrsv_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/,
      value /*const hipDoubleComplex **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/);
  // End
}
