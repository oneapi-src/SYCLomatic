#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, float *value, const int *row_ptr,
          const int *col_idx, csrsv2Info_t info) {
  // Start
  int buffer_size_in_bytes;
  hipsparseScsrsv2_bufferSize(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*float **/,
      row_ptr /*const int **/, col_idx /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size_in_bytes /*int **/);
  // End
}
