
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseDnVecDescr_t desc) {
  // Start
  int64_t size;
  void *value;
  hipDataType value_type;
  hipsparseDnVecGet(desc /*cusparseDnVecDescr_t*/, &size /*int64_t **/,
                   &value /*void ***/, &value_type /*cudaDataType **/);
  // End
}
