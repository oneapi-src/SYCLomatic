#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int k, int nnz, const double *alpha, const hipsparseMatDescr_t desc,
          const double *value, const int *row_ptr, const int *col_idx,
          const double *B, int ldb, const double *beta, double *C, int ldc) {
  // Start
  hipsparseDcsrmm(handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/,
                 m /*int*/, n /*int*/, k /*int*/, nnz /*int*/,
                 alpha /*const double **/, desc /*const hipsparseMatDescr_t*/,
                 value /*const double **/, row_ptr /*const int **/,
                 col_idx /*const int **/, B /*const double **/, ldb /*int*/,
                 beta /*const double **/, C /*double **/, ldc /*int*/);
  // End
}
