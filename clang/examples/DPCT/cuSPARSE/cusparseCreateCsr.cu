
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(int64_t rows, int64_t cols, int64_t nnz, void *row_ptr, void *col_ind,
          void *value, hipsparseIndexType_t row_ptr_type,
          hipsparseIndexType_t col_ind_type, hipsparseIndexBase_t base,
          hipDataType value_type) {
  // Start
  hipsparseSpMatDescr_t desc;
  hipsparseCreateCsr(&desc /*cusparseSpMatDescr_t **/, rows /*int64_t*/,
                    cols /*int64_t*/, nnz /*int64_t*/, row_ptr /*void **/,
                    col_ind /*void **/, value /*void **/,
                    row_ptr_type /*cusparseIndexType_t*/,
                    col_ind_type /*cusparseIndexType_t*/,
                    base /*cusparseIndexBase_t*/, value_type /*cudaDataType*/);
  // End
}
