#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int nnz, const hipDoubleComplex *alpha, hipsparseMatDescr_t desc,
          const hipDoubleComplex *value, const int *row_ptr, const int *col_idx,
          const hipDoubleComplex *x, const hipDoubleComplex *beta,
          hipDoubleComplex *y) {
  // Start
  cusparseZcsrmv_mp(handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/,
                    m /*int*/, n /*int*/, nnz /*int*/,
                    alpha /*const hipDoubleComplex **/,
                    desc /*hipsparseMatDescr_t*/,
                    value /*const hipDoubleComplex **/, row_ptr /*const int **/,
                    col_idx /*const int **/, x /*const hipDoubleComplex **/,
                    beta /*const hipDoubleComplex **/, y /*hipDoubleComplex **/);
  // End
}
