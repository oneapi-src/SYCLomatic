#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans_a,
          hipsparseOperation_t trans_b, int m, int n, int k, int nnz,
          const float *alpha, const hipsparseMatDescr_t desc, const float *value,
          const int *row_ptr, const int *col_idx, const float *B, int ldb,
          const float *beta, float *C, int ldc) {
  // Start
  hipsparseScsrmm2(handle /*hipsparseHandle_t*/, trans_a /*hipsparseOperation_t*/,
                  trans_b /*hipsparseOperation_t*/, m /*int*/, n /*int*/,
                  k /*int*/, nnz /*int*/, alpha /*const float **/,
                  desc /*const hipsparseMatDescr_t*/, value /*const float **/,
                  row_ptr /*const int **/, col_idx /*const int **/,
                  B /*const float **/, ldb /*int*/, beta /*const float **/,
                  C /*float **/, ldc /*int*/);
  // End
}
