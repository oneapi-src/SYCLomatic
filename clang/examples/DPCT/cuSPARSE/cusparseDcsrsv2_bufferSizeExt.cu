#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const hipsparseMatDescr_t desc, double *value, const int *row_ptr,
          const int *con_ind, csrsv2Info_t info) {
  // Start
  size_t buffer_size;
  hipsparseDcsrsv2_bufferSizeExt(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*const hipsparseMatDescr_t*/, value /*double **/,
      row_ptr /*const int **/, con_ind /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size /*size_t **/);
  // End
}
