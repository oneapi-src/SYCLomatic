
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t op_a, const void *alpha,
          hipsparseSpMatDescr_t mat_a, hipsparseDnVecDescr_t vec_x,
          hipsparseDnVecDescr_t vec_y, hipDataType compute_type,
          hipsparseSpSVAlg_t alg, hipsparseSpSVDescr_t desc) {
  // Start
  hipsparseSpSV_solve(handle /*cusparseHandle_t*/, op_a /*cusparseOperation_t*/,
                     alpha /*const void **/, mat_a /*cusparseSpMatDescr_t*/,
                     vec_x /*cusparseDnVecDescr_t*/,
                     vec_y /*cusparseDnVecDescr_t*/,
                     compute_type /*cudaDataType*/, alg /*cusparseSpSVAlg_t*/,
                     desc /*cusparseSpSVDescr_t*/);
  // End
}
