#include "hipsparse.h"

void test(hipsparseHandle_t handle, int m, int n, int nnz,
          const hipComplex *csr_value, const int *row_ptr, const int *col_idx,
          hipComplex *csc_value, int *row_ind, int *col_ptr,
          hipsparseAction_t act, hipsparseIndexBase_t base) {
  // Start
  hipsparseCcsr2csc(handle /*hipsparseHandle_t*/, m /*int*/, n /*int*/,
                   nnz /*int*/, csr_value /*const hipComplex **/,
                   row_ptr /*const int **/, col_idx /*const int **/,
                   csc_value /*hipComplex **/, row_ind /*int **/,
                   col_ptr /*int **/, act /*hipsparseAction_t*/,
                   base /*hipsparseIndexBase_t*/);
  // End
}
