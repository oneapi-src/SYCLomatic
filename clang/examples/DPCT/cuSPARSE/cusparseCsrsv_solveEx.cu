#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m,
          const void *alpha, hipDataType alpha_type,
          const hipsparseMatDescr_t desc, const void *value,
          hipDataType value_type, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, const void *f, hipDataType f_type,
          void *x, hipDataType x_type, hipDataType exec_type) {
  // Start
  cusparseCsrsv_solveEx(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      alpha /*const void **/, alpha_type /*hipDataType*/,
      desc /*const hipsparseMatDescr_t*/, value /*const void **/,
      value_type /*hipDataType*/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/,
      f /*const void **/, f_type /*hipDataType*/, x /*void **/,
      x_type /*hipDataType*/, exec_type /*hipDataType*/);
  // End
}
