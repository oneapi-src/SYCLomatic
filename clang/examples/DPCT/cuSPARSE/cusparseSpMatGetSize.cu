
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseSpMatDescr_t desc) {
  // Start
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  hipsparseSpMatGetSize(desc /*cusparseSpMatDescr_t*/, &rows /*int64_t **/,
                       &cols /*int64_t **/, &nnz /*int64_t **/);
  // End
}
