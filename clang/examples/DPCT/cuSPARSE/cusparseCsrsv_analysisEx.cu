#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const hipsparseMatDescr_t desc, const void *value,
          hipDataType value_type, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, hipDataType exec_type) {
  // Start
  cusparseCsrsv_analysisEx(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*const hipsparseMatDescr_t*/, value /*const void **/,
      value_type /*hipDataType*/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/,
      exec_type /*hipDataType*/);
  // End
}
