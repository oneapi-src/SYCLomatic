
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t transa,
          hipsparseOperation_t transb, const void *alpha,
          hipsparseConstSpMatDescr_t a, hipsparseConstDnMatDescr_t b,
          const void *beta, hipsparseDnMatDescr_t c, hipDataType computetype,
          hipsparseSpMMAlg_t algo, void *workspace) {
  // Start
  hipsparseSpMM(handle /*cusparseHandle_t*/, transa /*cusparseOperation_t*/,
               transb /*cusparseOperation_t*/, alpha /*const void **/,
               a /*cusparseConstSpMatDescr_t*/, b /*cusparseConstDnMatDescr_t*/,
               beta /*const void **/, c /*cusparseDnMatDescr_t*/,
               computetype /*cudaDataType*/, algo /*cusparseSpMMAlg_t*/,
               workspace /*void **/);
  // End
}
