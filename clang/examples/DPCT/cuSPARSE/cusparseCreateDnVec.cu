
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(int64_t size, void *value, hipDataType value_type) {
  // Start
  hipsparseDnVecDescr_t desc;
  hipsparseCreateDnVec(&desc /*cusparseDnVecDescr_t **/, size /*int64_t*/,
                      value /*void **/, value_type /*cudaDataType*/);
  // End
}
