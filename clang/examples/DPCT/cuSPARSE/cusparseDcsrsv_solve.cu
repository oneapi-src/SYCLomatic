#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m,
          const double *alpha, const hipsparseMatDescr_t desc,
          const double *value, const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info, const double *f, double *x) {
  // Start
  cusparseDcsrsv_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      alpha /*const double **/, desc /*const hipsparseMatDescr_t*/,
      value /*const double **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*cusparseSolveAnalysisInfo_t*/,
      f /*const double **/, x /*double **/);
  // End
}
