#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, const double *value, const int *row_ptr,
          const int *col_idx, csrsv2Info_t info, hipsparseSolvePolicy_t policy,
          void *buffer) {
  // Start
  hipsparseDcsrsv2_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*const double **/,
      row_ptr /*const int **/, col_idx /*const int **/, info /*csrsv2Info_t*/,
      policy /*hipsparseSolvePolicy_t*/, buffer /*void **/);
  // End
}
