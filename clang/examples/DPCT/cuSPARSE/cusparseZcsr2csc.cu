#include "hipsparse.h"

void test(hipsparseHandle_t handle, int m, int n, int nnz,
          const hipDoubleComplex *csr_value, const int *row_ptr,
          const int *col_idx, hipDoubleComplex *csc_value, int *row_ind,
          int *col_ptr, hipsparseAction_t act, hipsparseIndexBase_t base) {
  // Start
  hipsparseZcsr2csc(handle /*hipsparseHandle_t*/, m /*int*/, n /*int*/,
                   nnz /*int*/, csr_value /*const hipDoubleComplex **/,
                   row_ptr /*const int **/, col_idx /*const int **/,
                   csc_value /*hipDoubleComplex **/, row_ind /*int **/,
                   col_ptr /*int **/, act /*hipsparseAction_t*/,
                   base /*hipsparseIndexBase_t*/);
  // End
}
