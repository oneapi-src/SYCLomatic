#include "hipsparse.h"

void test(hipsparseHandle_t handle, int m, int n, int nnz,
          const double *csr_value, const int *row_ptr, const int *col_idx,
          double *csc_value, int *row_ind, int *col_ptr, hipsparseAction_t act,
          hipsparseIndexBase_t base) {
  // Start
  hipsparseDcsr2csc(handle /*hipsparseHandle_t*/, m /*int*/, n /*int*/,
                   nnz /*int*/, csr_value /*const double **/,
                   row_ptr /*const int **/, col_idx /*const int **/,
                   csc_value /*double **/, row_ind /*int **/, col_ptr /*int **/,
                   act /*hipsparseAction_t*/, base /*hipsparseIndexBase_t*/);
  // End
}
