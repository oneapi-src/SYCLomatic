#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, hipDoubleComplex *value, const int *row_ptr,
          const int *con_ind, csrsv2Info_t info) {
  // Start
  size_t buffer_size;
  hipsparseZcsrsv2_bufferSizeExt(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*hipDoubleComplex **/,
      row_ptr /*const int **/, con_ind /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size /*size_t **/);
  // End
}
