#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, double *value, const int *row_ptr,
          const int *col_idx, csrsv2Info_t info) {
  // Start
  int buffer_size_in_bytes;
  hipsparseDcsrsv2_bufferSize(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*double **/,
      row_ptr /*const int **/, col_idx /*const int **/, info /*csrsv2Info_t*/,
      &buffer_size_in_bytes /*int **/);
  // End
}
