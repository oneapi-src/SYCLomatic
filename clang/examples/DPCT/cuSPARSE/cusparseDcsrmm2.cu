#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans_a,
          hipsparseOperation_t trans_b, int m, int n, int k, int nnz,
          const double *alpha, const hipsparseMatDescr_t desc,
          const double *value, const int *row_ptr, const int *col_idx,
          const double *B, int ldb, const double *beta, double *C, int ldc) {
  // Start
  hipsparseDcsrmm2(handle /*hipsparseHandle_t*/, trans_a /*hipsparseOperation_t*/,
                  trans_b /*hipsparseOperation_t*/, m /*int*/, n /*int*/,
                  k /*int*/, nnz /*int*/, alpha /*const double **/,
                  desc /*const hipsparseMatDescr_t*/, value /*const double **/,
                  row_ptr /*const int **/, col_idx /*const int **/,
                  B /*const double **/, ldb /*int*/, beta /*const double **/,
                  C /*double **/, ldc /*int*/);
  // End
}
