
#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseSpMatDescr_t desc) {
  // Start
  int64_t rows;
  int64_t cols;
  int64_t nnz;
  void *row_ptr;
  void *col_ind;
  void *value;
  hipsparseIndexType_t row_ptr_type;
  hipsparseIndexType_t col_ind_type;
  hipsparseIndexBase_t base;
  hipDataType value_type;
  hipsparseCsrGet(
      desc /*cusparseSpMatDescr_t*/, &rows /*int64_t **/, &cols /*int64_t **/,
      &nnz /*int64_t **/, &row_ptr /*void ***/, &col_ind /*void ***/,
      &value /*void ***/, &row_ptr_type /*cusparseIndexType_t **/,
      &col_ind_type /*cusparseIndexType_t **/, &base /*cusparseIndexBase_t **/,
      &value_type /*cudaDataType **/);
  // End
}
