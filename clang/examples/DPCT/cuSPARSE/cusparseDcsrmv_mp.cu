#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int nnz, const double *alpha, const hipsparseMatDescr_t desc,
          const double *value, const int *row_ptr, const int *col_idx,
          const double *x, const double *beta, double *y) {
  // Start
  cusparseDcsrmv_mp(handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/,
                    m /*int*/, n /*int*/, nnz /*int*/, alpha /*const double **/,
                    desc /*const hipsparseMatDescr_t*/, value /*const double **/,
                    row_ptr /*const int **/, col_idx /*const int **/,
                    x /*const double **/, beta /*const double **/,
                    y /*double **/);
  // End
}
