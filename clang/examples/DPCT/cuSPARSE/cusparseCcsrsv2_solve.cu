#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const hipComplex *alpha, hipsparseMatDescr_t desc,
          const hipComplex *value, const int *row_ptr, const int *col_idx,
          csrsv2Info_t info, const hipComplex *f, hipComplex *x,
          hipsparseSolvePolicy_t policy, void *buffer) {
  // Start
  hipsparseCcsrsv2_solve(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, alpha /*const hipComplex **/, desc /*hipsparseMatDescr_t*/,
      value /*const hipComplex **/, row_ptr /*const int **/,
      col_idx /*const int **/, info /*csrsv2Info_t*/, f /*const hipComplex **/,
      x /*hipComplex **/, policy /*hipsparseSolvePolicy_t*/, buffer /*void **/);
  // End
}
