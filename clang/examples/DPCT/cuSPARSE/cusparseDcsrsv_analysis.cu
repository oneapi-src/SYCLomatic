#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          const hipsparseMatDescr_t desc, const double *value,
          const int *row_ptr, const int *col_idx,
          cusparseSolveAnalysisInfo_t info) {
  // Start
  cusparseDcsrsv_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*const hipsparseMatDescr_t*/, value /*const double **/,
      row_ptr /*const int **/, col_idx /*const int **/,
      info /*cusparseSolveAnalysisInfo_t*/);
  // End
}
