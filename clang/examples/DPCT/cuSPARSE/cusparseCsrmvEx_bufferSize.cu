#include "hipsparse.h"

void test(hipsparseHandle_t handle, cusparseAlgMode_t algo,
          hipsparseOperation_t trans, int m, int n, int nnz, const void *alpha,
          hipDataType alpha_type, hipsparseMatDescr_t desc, const void *value,
          hipDataType value_type, const int *row_ptr, const int *col_idx,
          const void *x, hipDataType x_type, const void *beta,
          hipDataType beta_type, void *y, hipDataType y_type,
          hipDataType exec_type) {
  // Start
  size_t buffer_size_in_bytes;
  cusparseCsrmvEx_bufferSize(
      handle /*hipsparseHandle_t*/, algo /*cusparseAlgMode_t*/,
      trans /*hipsparseOperation_t*/, m /*int*/, n /*int*/, nnz /*int*/,
      alpha /*const void **/, alpha_type /*hipDataType*/,
      desc /*hipsparseMatDescr_t*/, value /*const void **/,
      value_type /*hipDataType*/, row_ptr /*const int **/,
      col_idx /*const int **/, x /*const void **/, x_type /*hipDataType*/,
      beta /*const void **/, beta_type /*hipDataType*/, y /*void **/,
      y_type /*hipDataType*/, exec_type /*hipDataType*/,
      &buffer_size_in_bytes /*size_t **/);
  // End
}
