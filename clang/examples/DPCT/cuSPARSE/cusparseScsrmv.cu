#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int n,
          int nnz, const float *alpha, const hipsparseMatDescr_t desc,
          const float *value, const int *row_ptr, const int *col_idx,
          const float *x, const float *beta, float *y) {
  // Start
  hipsparseScsrmv(handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/,
                 m /*int*/, n /*int*/, nnz /*int*/, alpha /*const float **/,
                 desc /*const hipsparseMatDescr_t*/, value /*const float **/,
                 row_ptr /*const int **/, col_idx /*const int **/,
                 x /*const float **/, beta /*const float **/, y /*float **/);
  // End
}
