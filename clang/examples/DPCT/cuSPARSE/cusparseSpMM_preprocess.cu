// Migration desc: The API is Removed.

#include <hip/hip_runtime.h>
#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t transa,
          hipsparseOperation_t transb, const void *alpha, hipsparseSpMatDescr_t a,
          hipsparseDnMatDescr_t b, const void *beta, hipsparseDnMatDescr_t c,
          hipDataType computetype, hipsparseSpMMAlg_t algo, void *workspace) {
  // Start
  hipsparseSpMM_preprocess(
      handle /*cusparseHandle_t*/, transa /*cusparseOperation_t*/,
      transb /*cusparseOperation_t*/, alpha /*const void **/,
      a /*cusparseSpMatDescr_t*/, b /*cusparseDnMatDescr_t*/,
      beta /*const void **/, c /*cusparseDnMatDescr_t*/,
      computetype /*cudaDataType*/, algo /*cusparseSpMMAlg_t*/,
      workspace /*void **/);
  // End
}
