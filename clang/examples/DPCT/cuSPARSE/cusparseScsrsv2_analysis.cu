#include "hipsparse.h"

void test(hipsparseHandle_t handle, hipsparseOperation_t trans, int m, int nnz,
          hipsparseMatDescr_t desc, const float *value, const int *row_ptr,
          const int *col_idx, csrsv2Info_t info, hipsparseSolvePolicy_t policy,
          void *buffer) {
  // Start
  hipsparseScsrsv2_analysis(
      handle /*hipsparseHandle_t*/, trans /*hipsparseOperation_t*/, m /*int*/,
      nnz /*int*/, desc /*hipsparseMatDescr_t*/, value /*const float **/,
      row_ptr /*const int **/, col_idx /*const int **/, info /*csrsv2Info_t*/,
      policy /*hipsparseSolvePolicy_t*/, buffer /*void **/);
  // End
}
