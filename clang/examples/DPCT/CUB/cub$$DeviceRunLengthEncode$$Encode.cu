// clang-format off
#include <cstddef>
#include <hipcub/hipcub.hpp>

void test(void *temp_storage, size_t &temp_storage_bytes, int *d_in, int *d_unique_out, int *d_counts_out,int *d_num_runs_out, int num_items) {
  // Start
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipcub::DeviceRunLengthEncode::Encode(temp_storage/*void **/, temp_storage_bytes/*size_t &*/, d_in/*InputIteratorT*/, d_unique_out/*UniqueOutputIteratorT*/, d_counts_out/*LengthsOutputIteratorT*/, d_num_runs_out/*NumRunsOutputIteratorT*/, num_items/*int*/, stream/*hipStream_t*/);
  // End
}
