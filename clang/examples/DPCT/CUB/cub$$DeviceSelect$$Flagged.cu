// clang-format off
#include <cstddef>
#include <hipcub/hipcub.hpp>

void test(void *temp_storage, size_t &temp_storage_bytes, int *d_in, int *d_flags, int *d_out,int *d_num_selected_out,
          int num_items) {
  // Start
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipcub::DeviceSelect::Flagged(temp_storage/*void **/, temp_storage_bytes/*size_t &*/, d_in/*InputIteratorT*/, d_flags/*FlagIterator*/, d_out/*OutputIteratorT*/, d_num_selected_out/*NumSelectedIteratorT*/, num_items/*int*/, stream/*hipStream_t*/);
  // End
}
