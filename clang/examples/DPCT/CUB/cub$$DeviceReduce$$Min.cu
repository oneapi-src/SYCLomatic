// clang-format off
#include <cstddef>
#include <hipcub/hipcub.hpp>

void test(void *temp_storage, size_t &temp_storage_bytes, int *d_in, int *d_out, int num_items) {
  // Start
  hipStream_t stream;
  hipStreamCreate(&stream);
  hipcub::DeviceReduce::Min(temp_storage/*void **/, temp_storage_bytes/*size_t &*/, d_in/*InputIteratorT*/, d_out/*OutputIteratorT*/, num_items/*int*/, stream/*hipStream_t*/);
  // End
}
