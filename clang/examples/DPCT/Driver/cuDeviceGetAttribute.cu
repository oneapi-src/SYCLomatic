
#include <hip/hip_runtime.h>
void test(int *pi, hipDevice_t d) {
  // Start
  /* 1 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeMaxThreadsPerBlock, d /*CUdevice*/);
  /* 2 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeMaxBlockDimX, d /*CUdevice*/);
  /* 3 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeMaxBlockDimY, d /*CUdevice*/);
  /* 4 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeMaxBlockDimZ, d /*CUdevice*/);
  /* 5 */ hipDeviceGetAttribute(pi /*int **/,
                               hipDeviceAttributeMaxSharedMemoryPerBlock,
                               d /*CUdevice*/);
  /* 6 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeTotalConstantMemory, d /*CUdevice*/);
  /* 7 */ hipDeviceGetAttribute(pi /*int **/, hipDeviceAttributeWarpSize,
                               d /*CUdevice*/);
  /* 8 */ hipDeviceGetAttribute(pi /*int **/,
                               hipDeviceAttributeMaxRegistersPerBlock,
                               d /*CUdevice*/);
  /* 9 */ hipDeviceGetAttribute(pi /*int **/, hipDeviceAttributeClockRate,
                               d /*CUdevice*/);
  /* 10 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeTextureAlignment, d /*CUdevice*/);
  /* 11 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeMultiprocessorCount, d /*CUdevice*/);
  /* 12 */ hipDeviceGetAttribute(pi /*int **/, hipDeviceAttributeIntegrated,
                                d /*CUdevice*/);
  /* 13 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeCanMapHostMemory, d /*CUdevice*/);
  /* 14 */ hipDeviceGetAttribute(pi /*int **/,
                                hipDeviceAttributeComputeCapabilityMajor,
                                d /*CUdevice*/);
  /* 15 */ hipDeviceGetAttribute(pi /*int **/,
                                hipDeviceAttributeComputeCapabilityMinor,
                                d /*CUdevice*/);
  /* 16 */ hipDeviceGetAttribute(
      pi /*int **/, hipDeviceAttributeHostNativeAtomicSupported,
      d /*CUdevice*/);
  // End
}
