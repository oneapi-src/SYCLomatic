
#include <hip/hip_runtime.h>
void test(size_t s1, const void *pv, size_t s2, hipStream_t s) {
  // Start
  hipArray_t a;
  hipMemcpyHtoAAsync(a /*CUarray*/, s1 /*size_t*/, pv /*const void **/,
                    s2 /*size_t*/, s /*CUstream*/);
  // End
}
