
#include <hip/hip_runtime.h>
void test(hipArray_t a, size_t s1, const void *pv, size_t s2, hipStream_t s) {
  // Start
  hipMemcpyHtoAAsync(a /*CUarray*/, s1 /*size_t*/, pv /*const void **/,
                    s2 /*size_t*/, s /*CUstream*/);
  // End
}
