
#include <hip/hip_runtime.h>
void test(hipDeviceptr_t pd1, hipCtx_t c1, hipDeviceptr_t pd2, hipCtx_t c2,
          size_t s, hipStream_t cs) {
  // Start
  cuMemcpyPeerAsync(pd1 /*CUdeviceptr*/, c1 /*CUcontext*/, pd2 /*CUdeviceptr*/,
                    c2 /*CUcontext*/, s /*size_t*/, cs /*CUstream*/);
  // End
}
