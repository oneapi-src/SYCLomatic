
#include <hip/hip_runtime.h>
void test(hipFunction_t f, unsigned int u1, unsigned int u2, unsigned int u3,
          unsigned int u4, unsigned int u5, unsigned int u6, unsigned int u7,
          hipStream_t s, void **ppv1, void **ppv2) {
  // Start
  hipModuleLaunchKernel(f /*CUfunction*/, u1 /*unsigned int*/, u2 /*unsigned int*/,
                 u3 /*unsigned int*/, u4 /*unsigned int*/, u5 /*unsigned int*/,
                 u6 /*unsigned int*/, u7 /*unsigned int*/, s /*CUstream*/,
                 ppv1 /*void ***/, ppv2 /*void ***/);
  // End
}
