
#include <hip/hip_runtime.h>
void test(hipTextureObject_t *pt, const HIP_RESOURCE_DESC *pr,
          const HIP_TEXTURE_DESC *ptd, const HIP_RESOURCE_VIEW_DESC *prv) {
  // Start
  hipTexObjectCreate(pt /*CUtexObject **/, pr /*const CUDA_RESOURCE_DESC **/,
                    ptd /*const CUDA_TEXTURE_DESC **/,
                    prv /*const CUDA_RESOURCE_VIEW_DESC **/);
  // End
}
