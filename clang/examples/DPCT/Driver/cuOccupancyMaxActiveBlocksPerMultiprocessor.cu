
#include <hip/hip_runtime.h>
// Option: --use-experimental-features=occupancy-calculation

void test(int *pi, hipFunction_t f, int i, size_t s) {
  // Start
  hipModuleOccupancyMaxActiveBlocksPerMultiprocessor(pi /*int **/, f /*CUfunction*/,
                                              i /*int*/, s /*size_t*/);
  // End
}
