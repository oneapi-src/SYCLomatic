
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void test(hipDoubleComplex c1, hipDoubleComplex c2,
                     hipDoubleComplex c3) {
  // Start
  hipCfma(c1 /*cuDoubleComplex*/, c2 /*cuDoubleComplex*/,
         c3 /*cuDoubleComplex*/);
  // End
}
