
#include <hip/hip_runtime.h>
#include <hip/hip_complex.h>

__global__ void test(hipDoubleComplex c1, hipDoubleComplex c2) {
  // Start
  hipCadd(c1 /*cuDoubleComplex*/, c2 /*cuDoubleComplex*/);
  // End
}
