
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(cudnnActivationDescriptor_t desc, void *alpha,
          void *beta, cudnnTensorDescriptor_t dst_d, void *dst,
          cudnnTensorDescriptor_t diff_dst_d, void *diff_dst,
          cudnnTensorDescriptor_t diff_src_d, void *diff_src,
          cudnnTensorDescriptor_t src_d, void *src) {
  // Start
  cudnnHandle_t h;
  cudnnCreate(&h /*cudnnHandle_t **/);
  cudnnActivationBackward(
      h /*cudnnHandle_t*/, desc /*cudnnActivationDescriptor_t*/,
      alpha /*void **/, dst_d /*cudnnTensorDescriptor_t*/, dst /*void **/,
      diff_dst_d /*cudnnTensorDescriptor_t*/, diff_dst /*void **/,
      src_d /*cudnnTensorDescriptor_t*/, src /*void **/, beta /*void **/,
      diff_src_d /*cudnnTensorDescriptor_t*/, diff_src /*void **/);
  // End
}
