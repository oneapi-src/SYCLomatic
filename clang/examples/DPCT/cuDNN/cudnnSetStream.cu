
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(hipStream_t s) {
  // Start
  cudnnHandle_t h;
  cudnnCreate(&h /*cudnnHandle_t **/);
  cudnnSetStream(h /*cudnnHandle_t*/, s /*cudaStream_t*/);
  // End
}
