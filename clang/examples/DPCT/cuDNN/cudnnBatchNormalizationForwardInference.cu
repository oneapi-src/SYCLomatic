
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(cudnnBatchNormMode_t m, void *alpha, void *beta,
          cudnnTensorDescriptor_t src_d, void *src,
          cudnnTensorDescriptor_t dst_d, void *dst, cudnnTensorDescriptor_t p_d,
          void *scale, void *bias, void *mean, void *var, double eps) {
  // Start
  cudnnHandle_t h;
  cudnnCreate(&h /*cudnnHandle_t **/);
  cudnnBatchNormalizationForwardInference(
      h /*cudnnHandle_t*/, m /*cudnnBatchNormMode_t*/, alpha /*void **/,
      beta /*void **/, src_d /*cudnnTensorDescriptor_t*/, src /*void **/,
      dst_d /*cudnnTensorDescriptor_t*/, dst /*void **/,
      p_d /*cudnnTensorDescriptor_t*/, scale /*void **/, bias /*void **/,
      mean /*void **/, var /*void **/, eps /*double*/);
  // End
}