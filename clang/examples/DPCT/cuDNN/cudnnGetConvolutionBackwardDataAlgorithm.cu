#include <hipDNN.h>

void test(hipdnnFilterDescriptor_t filter_d, hipdnnTensorDescriptor_t diff_dst_d,
          hipdnnConvolutionDescriptor_t cdesc,
          hipdnnTensorDescriptor_t diff_src_d,
          hipdnnConvolutionBwdDataPreference_t preference, size_t limit,
          hipdnnConvolutionBwdDataAlgo_t *alg) {
  // Start
  hipdnnHandle_t h;
  hipdnnCreate(&h /*hipdnnHandle_t **/);
  hipdnnGetConvolutionBackwardDataAlgorithm(
      h /*hipdnnHandle_t*/, filter_d /*hipdnnFilterDescriptor_t*/,
      diff_dst_d /*hipdnnTensorDescriptor_t*/,
      cdesc /*hipdnnConvolutionDescriptor_t*/,
      diff_src_d /*hipdnnTensorDescriptor_t*/,
      preference /*hipdnnConvolutionBwdDataPreference_t*/, limit /*size_t*/,
      alg /*hipdnnConvolutionBwdDataAlgo_t **/);
  // End
}