
#include <hip/hip_runtime.h>
#include <cudnn.h>

void test(cudnnNormMode_t m, cudnnNormOps_t op, cudnnNormAlgo_t alg,
          cudnnTensorDescriptor_t src_d, cudnnTensorDescriptor_t summand_d,
          cudnnTensorDescriptor_t dst_d, cudnnTensorDescriptor_t p1_d,
          cudnnActivationDescriptor_t adesc, cudnnTensorDescriptor_t p2_d,
          size_t *size, int group_count) {
  // Start
  cudnnHandle_t h;
  cudnnCreate(&h /*cudnnHandle_t **/);
  cudnnGetNormalizationForwardTrainingWorkspaceSize(
      h /*cudnnHandle_t*/, m /*cudnnNormMode_t*/, op /*cudnnNormOps_t*/,
      alg /*cudnnNormAlgo_t*/, src_d /*cudnnTensorDescriptor_t*/,
      summand_d /*cudnnTensorDescriptor_t*/, dst_d /*cudnnTensorDescriptor_t*/,
      p1_d /*cudnnTensorDescriptor_t*/, adesc /*cudnnActivationDescriptor_t*/,
      p2_d /*cudnnTensorDescriptor_t*/, size /*size_t **/, group_count /*int*/);
  // End
}