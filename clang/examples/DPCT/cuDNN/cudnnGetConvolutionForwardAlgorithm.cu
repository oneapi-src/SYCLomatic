#include <hipDNN.h>

void test(hipdnnTensorDescriptor_t src_d, hipdnnFilterDescriptor_t filter_d,
          hipdnnConvolutionDescriptor_t cdesc, hipdnnTensorDescriptor_t dst_d,
          hipdnnConvolutionFwdPreference_t preference, size_t limit,
          hipdnnConvolutionFwdAlgo_t *alg) {
  // Start
  hipdnnHandle_t h;
  hipdnnCreate(&h /*hipdnnHandle_t **/);
  hipdnnGetConvolutionForwardAlgorithm(
      h /*hipdnnHandle_t*/, src_d /*hipdnnTensorDescriptor_t*/,
      filter_d /*hipdnnFilterDescriptor_t*/,
      cdesc /*hipdnnConvolutionDescriptor_t*/, dst_d /*hipdnnTensorDescriptor_t*/,
      preference /*hipdnnConvolutionFwdPreference_t*/, limit /*size_t*/,
      alg /*hipdnnConvolutionFwdAlgo_t **/);
  // End
}