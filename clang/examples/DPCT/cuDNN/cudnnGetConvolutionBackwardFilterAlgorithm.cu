#include <hipDNN.h>

void test(hipdnnTensorDescriptor_t src_d, hipdnnTensorDescriptor_t diff_dst_d,
          hipdnnConvolutionDescriptor_t cdesc,
          hipdnnFilterDescriptor_t diff_filter_d,
          hipdnnConvolutionBwdFilterPreference_t preference, size_t limit,
          hipdnnConvolutionBwdFilterAlgo_t *alg) {
  // Start
  hipdnnHandle_t h;
  hipdnnCreate(&h /*hipdnnHandle_t **/);
  hipdnnGetConvolutionBackwardFilterAlgorithm(
      h /*hipdnnHandle_t*/, src_d /*hipdnnTensorDescriptor_t*/,
      diff_dst_d /*hipdnnTensorDescriptor_t*/,
      cdesc /*hipdnnConvolutionDescriptor_t*/,
      diff_filter_d /*hipdnnFilterDescriptor_t*/,
      preference /*hipdnnConvolutionBwdFilterPreference_t*/, limit /*size_t*/,
      alg /*hipdnnConvolutionBwdFilterAlgo_t **/);
  // End
}