
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(float f1, float f2) {
  // Start
  hiprandStateMRG32k3a_t *ps1;
  hiprandStatePhilox4_32_10_t *ps2;
  hiprandStateXORWOW_t *ps3;
  hiprand_log_normal(ps1 /*curandStateMRG32k3a_t **/, f1 /*float*/,
                    f2 /*float*/);
  hiprand_log_normal(ps2 /*curandStatePhilox4_32_10_t **/, f1 /*float*/,
                    f2 /*float*/);
  hiprand_log_normal(ps3 /*curandStateXORWOW_t **/, f1 /*float*/, f2 /*float*/);
  // End
}
