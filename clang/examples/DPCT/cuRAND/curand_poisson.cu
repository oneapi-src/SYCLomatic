
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(double d) {
  // Start
  hiprandStateMRG32k3a_t *ps1;
  hiprandStatePhilox4_32_10_t *ps2;
  hiprandStateXORWOW_t *ps3;
  hiprand_poisson(ps1 /*curandStateMRG32k3a_t **/, d /*double*/);
  hiprand_poisson(ps2 /*curandStatePhilox4_32_10_t **/, d /*double*/);
  hiprand_poisson(ps3 /*curandStateXORWOW_t **/, d /*double*/);
  // End
}
