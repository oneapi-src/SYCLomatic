
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(double d) {
  // Start
  hiprandStateScrambledSobol64_t *ps1;
  hiprandStateSobol64_t *ps2;
  hiprandStateScrambledSobol32_t *ps3;
  hiprandStateSobol32_t *ps4;
  hiprandStateMtgp32_t *ps5;
  hiprandStateMRG32k3a_t *ps6;
  hiprandStatePhilox4_32_10_t *ps7;
  hiprandStateXORWOW_t *ps8;
  hiprand_poisson(ps1 /*curandStateScrambledSobol64_t **/, d /*double*/);
  hiprand_poisson(ps2 /*curandStateSobol64_t **/, d /*double*/);
  hiprand_poisson(ps3 /*curandStateScrambledSobol32_t **/, d /*double*/);
  hiprand_poisson(ps4 /*curandStateSobol32_t **/, d /*double*/);
  hiprand_poisson(ps5 /*curandStateMtgp32_t **/, d /*double*/);
  hiprand_poisson(ps6 /*curandStateMRG32k3a_t **/, d /*double*/);
  hiprand_poisson(ps7 /*curandStatePhilox4_32_10_t **/, d /*double*/);
  hiprand_poisson(ps8 /*curandStateXORWOW_t **/, d /*double*/);
  // End
}
