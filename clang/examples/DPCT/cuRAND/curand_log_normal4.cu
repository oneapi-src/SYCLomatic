
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(float f1, float f2) {
  // Start
  hiprandStatePhilox4_32_10_t *ps;
  hiprand_log_normal4(ps /*curandStatePhilox4_32_10_t **/, f1 /*float*/,
                     f2 /*float*/);
  // End
}
