
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(double d1, double d2) {
  // Start
  hiprandStateScrambledSobol64_t *ps1;
  hiprandStateSobol64_t *ps2;
  hiprandStateScrambledSobol32_t *ps3;
  hiprandStateSobol32_t *ps4;
  hiprandStateMtgp32_t *ps5;
  hiprandStateMRG32k3a_t *ps6;
  hiprandStatePhilox4_32_10_t *ps7;
  hiprandStateXORWOW_t *ps8;
  hiprand_log_normal_double(ps1 /*curandStateScrambledSobol64_t **/,
                           d1 /*double*/, d2 /*double*/);
  hiprand_log_normal_double(ps2 /*curandStateSobol64_t **/, d1 /*double*/,
                           d2 /*double*/);
  hiprand_log_normal_double(ps3 /*curandStateScrambledSobol32_t **/,
                           d1 /*double*/, d2 /*double*/);
  hiprand_log_normal_double(ps4 /*curandStateSobol32_t **/, d1 /*double*/,
                           d2 /*double*/);
  hiprand_log_normal_double(ps5 /*curandStateMtgp32_t **/, d1 /*double*/,
                           d2 /*double*/);
  hiprand_log_normal_double(ps6 /*curandStateMRG32k3a_t **/, d1 /*double*/,
                           d2 /*double*/);
  hiprand_log_normal_double(ps7 /*curandStatePhilox4_32_10_t **/, d1 /*double*/,
                           d2 /*double*/);
  hiprand_log_normal_double(ps8 /*curandStateXORWOW_t **/, d1 /*double*/,
                           d2 /*double*/);
  // End
}
