
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(unsigned long long ull1, unsigned long long ull2,
                     unsigned long long ull3, hiprandStateMRG32k3a_t *ps1,
                     hiprandStatePhilox4_32_10_t *ps2,
                     hiprandStateXORWOW_t *ps3) {
  // Start
  hiprand_init(ull1 /*unsigned long long*/, ull2 /*unsigned long long*/,
              ull3 /*unsigned long long*/, ps1 /*curandStateMRG32k3a_t **/);
  hiprand_init(ull1 /*unsigned long long*/, ull2 /*unsigned long long*/,
              ull3 /*unsigned long long*/,
              ps2 /*curandStatePhilox4_32_10_t **/);
  hiprand_init(ull1 /*unsigned long long*/, ull2 /*unsigned long long*/,
              ull3 /*unsigned long long*/, ps3 /*curandStateXORWOW_t **/);
  // End
}
