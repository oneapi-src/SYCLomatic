
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test() {
  // Start
  hiprandStateScrambledSobol64_t *ps1;
  hiprandStateSobol64_t *ps2;
  hiprandStateScrambledSobol32_t *ps3;
  hiprandStateSobol32_t *ps4;
  hiprandStateMtgp32_t *ps5;
  hiprandStateMRG32k3a_t *ps6;
  hiprandStatePhilox4_32_10_t *ps7;
  hiprandStateXORWOW_t *ps8;
  hiprand_uniform_double(ps1 /*curandStateScrambledSobol64_t **/);
  hiprand_uniform_double(ps2 /*curandStateSobol64_t **/);
  hiprand_uniform_double(ps3 /*curandStateScrambledSobol32_t **/);
  hiprand_uniform_double(ps4 /*curandStateSobol32_t **/);
  hiprand_uniform_double(ps5 /*curandStateMtgp32_t **/);
  hiprand_uniform_double(ps6 /*curandStateMRG32k3a_t **/);
  hiprand_uniform_double(ps7 /*curandStatePhilox4_32_10_t **/);
  hiprand_uniform_double(ps8 /*curandStateXORWOW_t **/);
  // End
}
