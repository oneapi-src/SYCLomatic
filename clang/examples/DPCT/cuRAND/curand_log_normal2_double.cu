
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(double d1, double d2) {
  // Start
  hiprandStateMRG32k3a_t *ps1;
  hiprandStatePhilox4_32_10_t *ps2;
  hiprandStateXORWOW_t *ps3;
  hiprand_log_normal2_double(ps1 /*curandStateMRG32k3a_t **/, d1 /*double*/,
                            d2 /*double*/);
  hiprand_log_normal2_double(ps2 /*curandStatePhilox4_32_10_t **/, d1 /*double*/,
                            d2 /*double*/);
  hiprand_log_normal2_double(ps3 /*curandStateXORWOW_t **/, d1 /*double*/,
                            d2 /*double*/);
  // End
}
