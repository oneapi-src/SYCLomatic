
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test() {
  // Start
  hiprandStateScrambledSobol64_t *ps1;
  hiprandStateSobol64_t *ps2;
  hiprandStateScrambledSobol32_t *ps3;
  hiprandStateSobol32_t *ps4;
  hiprandStateMtgp32_t *ps5;
  hiprandStateMRG32k3a_t *ps6;
  hiprandStatePhilox4_32_10_t *ps7;
  hiprandStateXORWOW_t *ps8;
  hiprand_normal(ps1 /*curandStateScrambledSobol64_t **/);
  hiprand_normal(ps2 /*curandStateSobol64_t **/);
  hiprand_normal(ps3 /*curandStateScrambledSobol32_t **/);
  hiprand_normal(ps4 /*curandStateSobol32_t **/);
  hiprand_normal(ps5 /*curandStateMtgp32_t **/);
  hiprand_normal(ps6 /*curandStateMRG32k3a_t **/);
  hiprand_normal(ps7 /*curandStatePhilox4_32_10_t **/);
  hiprand_normal(ps8 /*curandStateXORWOW_t **/);
  // End
}
