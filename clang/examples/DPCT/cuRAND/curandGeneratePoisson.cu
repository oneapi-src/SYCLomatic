
#include <hip/hip_runtime.h>
#include "hiprand/hiprand.h"

void test(unsigned int *pu, size_t s, double d) {
  // Start
  hiprandGenerator_t g;
  hiprandGeneratePoisson(g /*curandGenerator_t*/, pu /*unsigned int **/,
                        s /*size_t*/, d /*double*/);
  // End
}
