
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test() {
  // Start
  hiprandStateMRG32k3a_t *ps1;
  hiprandStatePhilox4_32_10_t *ps2;
  hiprandStateXORWOW_t *ps3;
  hiprand_normal2_double(ps1 /*curandStateMRG32k3a_t **/);
  hiprand_normal2_double(ps2 /*curandStatePhilox4_32_10_t **/);
  hiprand_normal2_double(ps3 /*curandStateXORWOW_t **/);
  // End
}
