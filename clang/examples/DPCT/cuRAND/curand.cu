
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test() {
  // Start
  hiprandStateMtgp32_t *ps1;
  hiprandStateScrambledSobol64_t *ps2;
  hiprandStateSobol64_t *ps3;
  hiprandStateScrambledSobol32_t *ps4;
  hiprandStateSobol32_t *ps5;
  hiprandStateMRG32k3a_t *ps6;
  hiprandStatePhilox4_32_10_t *ps7;
  hiprandStateXORWOW_t *ps8;
  hiprand(ps1 /*curandStateMtgp32_t **/);
  hiprand(ps2 /*curandStateScrambledSobol64_t **/);
  hiprand(ps3 /*curandStateSobol64_t **/);
  hiprand(ps4 /*curandStateScrambledSobol32_t **/);
  hiprand(ps5 /*curandStateSobol32_t **/);
  hiprand(ps6 /*curandStateMRG32k3a_t **/);
  hiprand(ps7 /*curandStatePhilox4_32_10_t **/);
  hiprand(ps8 /*curandStateXORWOW_t **/);
  // End
}
