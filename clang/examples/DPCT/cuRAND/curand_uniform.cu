
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test() {
  // Start
  hiprandStateMRG32k3a_t *ps1;
  hiprandStatePhilox4_32_10_t *ps2;
  hiprandStateXORWOW_t *ps3;
  hiprand_uniform(ps1 /*curandStateMRG32k3a_t **/);
  hiprand_uniform(ps2 /*curandStatePhilox4_32_10_t **/);
  hiprand_uniform(ps3 /*curandStateXORWOW_t **/);
  // End
}
