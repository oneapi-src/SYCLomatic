
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(unsigned long long ull) {
  // Start
  hiprandStateMRG32k3a_t *ps;
  skipahead_subsequence(ull, ps /*curandStateMRG32k3a_t **/);
  // End
}
