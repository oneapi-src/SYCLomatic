
#include <hip/hip_runtime.h>
#include "hiprand/hiprand_kernel.h"

__global__ void test(double d) {
  // Start
  hiprandStatePhilox4_32_10_t *ps;
  hiprand_poisson4(ps /*curandStatePhilox4_32_10_t **/, d /*double*/);
  // End
}
