
#include <hip/hip_runtime.h>
#include <nccl.h>

void test(const void *sendbuff, void *recvbuff, size_t count,
          ncclDataType_t datatype, int root, ncclComm_t comm,
          hipStream_t stream) {
  // Start
  ncclBroadcast(sendbuff /*void **/, recvbuff /*void **/, count /*size_t*/,
            datatype /*ncclDataType_t*/, root /*int*/, comm /*ncclComm_t*/,
            stream /*cudaStream_t*/);
  // End
}