
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const double *a, int lda,
          const double *w, hipsolverSyevjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnDsyevj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const double **/, lda /*int*/,
      w /*const double **/, &buffer_size /*int **/, params /*syevjInfo_t*/);
  // End
}
