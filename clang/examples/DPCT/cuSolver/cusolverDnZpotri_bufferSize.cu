
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, int *Lwork) {
  // Start
  int buffer_size;
  hipsolverDnZpotri_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*cuDoubleComplex **/, lda /*int*/, &buffer_size /*int **/);
  // End
}
