
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int n, hipDoubleComplex *a, int lda) {
  // Start
  int buffer_size;
  hipsolverDnZsytrf_bufferSize(handle /*cusolverDnHandle_t*/, n /*int*/,
                              a /*cuDoubleComplex **/, lda /*int*/,
                              &buffer_size /*int **/);
  // End
}
