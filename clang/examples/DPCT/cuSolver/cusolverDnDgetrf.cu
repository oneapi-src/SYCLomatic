
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, double *a, int lda,
          double *buffer, int *ipiv, int *info) {
  // Start
  hipsolverDnDgetrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*double **/, lda /*int*/, buffer /*double **/,
                   ipiv /*int **/, info /*int **/);
  // End
}
