
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, double *a, int lda) {
  // Start
  int buffer_size;
  hipsolverDnDgeqrf_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, a /*double **/, lda /*int*/,
                              &buffer_size /*int **/);
  // End
}
