
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const float *a,
          int lda, const float *b, int ldb, const float *w) {
  // Start
  int buffer_size;
  hipsolverDnSsygvd_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const float **/, lda /*int*/, b /*const float **/, ldb /*int*/,
      w /*const float **/, &buffer_size /*int **/);
  // End
}
