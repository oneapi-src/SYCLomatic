
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipDoubleComplex *a, int lda,
          double *d, double *e, hipDoubleComplex *tau_q, hipDoubleComplex *tau_p,
          hipDoubleComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnZgebrd(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuDoubleComplex **/, lda /*int*/, d /*double **/,
                   e /*double **/, tau_q /*cuDoubleComplex **/,
                   tau_p /*cuDoubleComplex **/, buffer /*cuDoubleComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
