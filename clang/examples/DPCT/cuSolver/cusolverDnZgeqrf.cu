
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipDoubleComplex *a, int lda,
          hipDoubleComplex *tau, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZgeqrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuDoubleComplex **/, lda /*int*/,
                   tau /*cuDoubleComplex **/, buffer /*cuDoubleComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
