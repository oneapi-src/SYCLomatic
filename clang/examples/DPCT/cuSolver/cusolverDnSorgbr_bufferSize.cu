
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          int k, const float *a, int lda, const float *tau) {
  // Start
  int buffer_size;
  hipsolverDnSorgbr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/, m /*int*/,
      n /*int*/, k /*int*/, a /*const float **/, lda /*int*/,
      tau /*const float **/, &buffer_size /*int **/);
  // End
}
