
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const hipDoubleComplex *a, int lda, const double *d, const double *e,
          const hipDoubleComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnZhetrd_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/, d /*const double **/,
      e /*const double **/, tau /*const cuDoubleComplex **/,
      &buffer_size /*int **/);
  // End
}
