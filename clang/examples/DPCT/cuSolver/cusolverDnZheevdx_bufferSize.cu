
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipsolverEigRange_t range, hipblasFillMode_t uplo, int n,
          const hipDoubleComplex *a, int lda, double vl, double vu, int il,
          int iu, int *h_meig, const double *w) {
  // Start
  int buffer_size;
  hipsolverDnZheevdx_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      range /*cusolverEigRange_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/, vl /*double*/, vu /*double*/,
      il /*int*/, iu /*int*/, h_meig /*int **/, w /*const double **/,
      &buffer_size /*int **/);
  // End
}
