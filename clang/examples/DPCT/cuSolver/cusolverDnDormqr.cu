
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k, const double *a,
          int lda, const double *tau, double *c, int ldc, double *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnDormqr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      trans /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const double **/, lda /*int*/, tau /*const double **/, c /*double **/,
      ldc /*int*/, buffer /*double **/, buffer_size /*int*/, info /*int **/);
  // End
}
