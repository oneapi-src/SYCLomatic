
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n,
          double *a, int lda, double *tau, double *c, int ldc, double *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnDormtr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      uplo /*cublasFillMode_t*/, trans /*cublasOperation_t*/, m /*int*/,
      n /*int*/, a /*double **/, lda /*int*/, tau /*double **/, c /*double **/,
      ldc /*int*/, buffer /*double **/, buffer_size /*int*/, info /*int **/);
  // End
}
