
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipDoubleComplex *a, int lda,
          hipDoubleComplex *buffer, int *ipiv, int *info) {
  // Start
  hipsolverDnZgetrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuDoubleComplex **/, lda /*int*/,
                   buffer /*cuDoubleComplex **/, ipiv /*int **/,
                   info /*int **/);
  // End
}
