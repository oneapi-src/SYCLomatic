
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, float *a, int lda,
          float *buffer, int *ipiv, int *info) {
  // Start
  hipsolverDnSgetrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*float **/, lda /*int*/, buffer /*float **/,
                   ipiv /*int **/, info /*int **/);
  // End
}
