
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs,
          const float *a, int lda, float *b, int ldb, int *info) {
  // Start
  hipsolverDnSpotrs(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, nrhs /*int*/, a /*const float **/, lda /*int*/,
                   b /*float **/, ldb /*int*/, info /*int **/);
  // End
}
