
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs,
          const hipComplex *a, int lda, hipComplex *b, int ldb, int *info) {
  // Start
  hipsolverDnCpotrs(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, nrhs /*int*/, a /*const cuComplex **/,
                   lda /*int*/, b /*cuComplex **/, ldb /*int*/, info /*int **/);
  // End
}
