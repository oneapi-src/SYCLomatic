
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params, int64_t m,
          int64_t n, hipDataType a_type, void *a, int64_t lda,
          hipDataType tau_type, void *tau, hipDataType compute_type,
          void *device_buffer, size_t device_buffer_size, void *host_buffer,
          size_t host_buffer_size, int *info) {
  // Start
  hipsolverDnXgeqrf(handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
                   m /*int64_t*/, n /*int64_t*/, a_type /*cudaDataType*/,
                   a /*void **/, lda /*int64_t*/, tau_type /*cudaDataType*/,
                   tau /*void **/, compute_type /*cudaDataType*/,
                   device_buffer /*void **/, device_buffer_size /*size_t*/,
                   host_buffer /*void **/, host_buffer_size /*size_t*/,
                   info /*int **/);
  // End
}
