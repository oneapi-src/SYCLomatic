#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipblasFillMode_t uplo, int64_t n, hipDataType a_type, const void *a,
          int64_t lda, hipDataType compute_type) {
  // Start
  size_t buffer_size;
  cusolverDnPotrf_bufferSize(
      handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
      uplo /*hipblasFillMode_t*/, n /*int64_t*/, a_type /*hipDataType*/,
      a /*const void **/, lda /*int64_t*/, compute_type /*hipDataType*/,
      &buffer_size /*size_t **/);
  // End
}
