
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const float *a, int lda, const float *tau) {
  // Start
  int buffer_size;
  hipsolverDnSorgtr_bufferSize(handle /*cusolverDnHandle_t*/,
                              uplo /*cublasFillMode_t*/, n /*int*/,
                              a /*const float **/, lda /*int*/,
                              tau /*const float **/, &buffer_size /*int **/);
  // End
}
