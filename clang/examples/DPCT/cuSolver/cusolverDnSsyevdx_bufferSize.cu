
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipsolverEigRange_t range, hipblasFillMode_t uplo, int n,
          const float *a, int lda, float vl, float vu, int il, int iu,
          int *h_meig, const float *w) {
  // Start
  int buffer_size;
  hipsolverDnSsyevdx_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      range /*cusolverEigRange_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const float **/, lda /*int*/, vl /*float*/, vu /*float*/, il /*int*/,
      iu /*int*/, h_meig /*int **/, w /*const float **/,
      &buffer_size /*int **/);
  // End
}
