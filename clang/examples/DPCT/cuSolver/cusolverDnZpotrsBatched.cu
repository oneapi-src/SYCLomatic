
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t upper_lower, int n,
          int nrhs, hipDoubleComplex **a, int lda, hipDoubleComplex **b, int ldb,
          int *info, int group_count) {
  // Start
  hipsolverDnZpotrsBatched(handle /*cusolverDnHandle_t*/,
                          upper_lower /*cublasFillMode_t*/, n /*int*/,
                          nrhs /*int*/, a /*cuDoubleComplex ***/, lda /*int*/,
                          b /*cuDoubleComplex ***/, ldb /*int*/, info /*int **/,
                          group_count /*int*/);
  // End
}
