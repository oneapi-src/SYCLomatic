
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, hipComplex *a, int lda, hipComplex *b,
          int ldb, float vl, float vu, int il, int iu, int *h_meig, float *w,
          hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnChegvdx(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                    jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
                    uplo /*cublasFillMode_t*/, n /*int*/, a /*cuComplex **/,
                    lda /*int*/, b /*cuComplex **/, ldb /*int*/, vl /*float*/,
                    vu /*float*/, il /*int*/, iu /*int*/, h_meig /*int **/,
                    w /*float **/, buffer /*cuComplex **/, buffer_size /*int*/,
                    info /*int **/);
  // End
}
