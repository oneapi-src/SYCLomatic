#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, hipDataType a_type, const void *a,
          int64_t lda, void *vl, void *vu, int64_t il, int64_t iu,
          int64_t *h_meig, hipDataType w_type, const void *w,
          hipDataType compute_type) {
  // Start
  size_t buffer_size;
  cusolverDnSyevdx_bufferSize(
      handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
      jobz /*hipsolverEigMode_t*/, range /*hipsolverEigRange_t*/,
      uplo /*hipblasFillMode_t*/, n /*int64_t*/, a_type /*hipDataType*/,
      a /*const void **/, lda /*int64_t*/, vl /*void **/, vu /*void **/,
      il /*int64_t*/, iu /*int64_t*/, h_meig /*int64_t **/,
      w_type /*hipDataType*/, w /*const void **/,
      compute_type /*hipDataType*/, &buffer_size /*size_t **/);
  // End
}
