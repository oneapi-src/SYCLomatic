
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, float *a, int lda, float *w,
          float *buffer, int buffer_size, int *info, hipsolverSyevjInfo_t params) {
  // Start
  hipsolverDnSsyevj(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                   uplo /*cublasFillMode_t*/, n /*int*/, a /*float **/,
                   lda /*int*/, w /*float **/, buffer /*float **/,
                   buffer_size /*int*/, info /*int **/, params /*syevjInfo_t*/);
  // End
}
