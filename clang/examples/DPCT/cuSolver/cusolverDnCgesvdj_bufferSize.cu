
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, const hipComplex *a, int lda, const float *s,
          const hipComplex *u, int ldu, const hipComplex *v, int ldv,
          hipsolverGesvdjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnCgesvdj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/, econ /*int*/,
      m /*int*/, n /*int*/, a /*const cuComplex **/, lda /*int*/,
      s /*const float **/, u /*const cuComplex **/, ldu /*int*/,
      v /*const cuComplex **/, ldv /*int*/, &buffer_size /*int **/,
      params /*gesvdjInfo_t*/);
  // End
}
