
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, const hipDoubleComplex *a, int lda, const double *s,
          const hipDoubleComplex *u, int ldu, const hipDoubleComplex *v, int ldv,
          hipsolverGesvdjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnZgesvdj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/, econ /*int*/,
      m /*int*/, n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
      s /*const double **/, u /*const cuDoubleComplex **/, ldu /*int*/,
      v /*const cuDoubleComplex **/, ldv /*int*/, &buffer_size /*int **/,
      params /*gesvdjInfo_t*/);
  // End
}
