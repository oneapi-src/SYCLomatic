
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, hipComplex *b, int ldb, float *w, hipComplex *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnChegvd(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuComplex **/, lda /*int*/, b /*cuComplex **/,
                   ldb /*int*/, w /*float **/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
