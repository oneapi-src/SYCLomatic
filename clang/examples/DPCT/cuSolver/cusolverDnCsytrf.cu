
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, int *ipiv, hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCsytrf(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuComplex **/, lda /*int*/, ipiv /*int **/,
                   buffer /*cuComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
