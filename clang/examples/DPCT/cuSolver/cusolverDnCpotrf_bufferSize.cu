
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda) {
  // Start
  int buffer_size;
  hipsolverDnCpotrf_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*cuComplex **/, lda /*int*/, &buffer_size /*int **/);
  // End
}
