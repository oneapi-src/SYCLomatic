
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, hipDoubleComplex *a,
          int lda, const hipDoubleComplex *tau, hipDoubleComplex *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnZungqr(
      handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*cuDoubleComplex **/, lda /*int*/, tau /*const cuDoubleComplex **/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
