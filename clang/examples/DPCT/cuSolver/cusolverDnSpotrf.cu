
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float *a,
          int lda, float *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnSpotrf(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*float **/, lda /*int*/, buffer /*float **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
