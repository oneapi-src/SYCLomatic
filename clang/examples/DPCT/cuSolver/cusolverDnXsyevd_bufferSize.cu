
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int64_t n,
          hipDataType a_type, const void *a, int64_t lda, hipDataType w_type,
          const void *w, hipDataType compute_type) {
  // Start
  size_t device_buffer_size;
  size_t host_buffer_size;
  cusolverDnXsyevd_bufferSize(
      handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int64_t*/,
      a_type /*cudaDataType*/, a /*const void **/, lda /*int64_t*/,
      w_type /*cudaDataType*/, w /*const void **/,
      compute_type /*cudaDataType*/, &device_buffer_size /*size_t **/,
      &host_buffer_size /*size_t **/);
  // End
}
