
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, float vl, float vu, int il, int iu, int *h_meig, float *w,
          hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCheevdx(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                    range /*cusolverEigRange_t*/, uplo /*cublasFillMode_t*/,
                    n /*int*/, a /*cuComplex **/, lda /*int*/, vl /*float*/,
                    vu /*float*/, il /*int*/, iu /*int*/, h_meig /*int **/,
                    w /*float **/, buffer /*cuComplex **/, buffer_size /*int*/,
                    info /*int **/);
  // End
}
