
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo,
          hipblasDiagType_t diag, int64_t n, hipDataType a_type, void *a,
          int64_t lda) {
  // Start
  size_t device_buffer_size;
  size_t host_buffer_size;
  cusolverDnXtrtri_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
      diag /*cublasDiagType_t*/, n /*int64_t*/, a_type /*cudaDataType*/,
      a /*void **/, lda /*int64_t*/, &device_buffer_size /*size_t **/,
      &host_buffer_size /*size_t **/);
  // End
}
