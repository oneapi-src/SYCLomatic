
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipsolverEigRange_t range, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, double vl, double vu, int il, int iu,
          int *h_meig, double *w, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZheevdx(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      range /*cusolverEigRange_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*cuDoubleComplex **/, lda /*int*/, vl /*double*/, vu /*double*/,
      il /*int*/, iu /*int*/, h_meig /*int **/, w /*double **/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
