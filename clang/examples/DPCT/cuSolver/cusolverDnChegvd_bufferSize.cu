
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          const float *w) {
  // Start
  int buffer_size;
  hipsolverDnChegvd_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuComplex **/, lda /*int*/, b /*const cuComplex **/,
      ldb /*int*/, w /*const float **/, &buffer_size /*int **/);
  // End
}
