
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, double *a, int lda, double *b, int ldb,
          double vl, double vu, int il, int iu, int *h_meig, double *w,
          double *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnDsygvdx(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                    jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
                    uplo /*cublasFillMode_t*/, n /*int*/, a /*double **/,
                    lda /*int*/, b /*double **/, ldb /*int*/, vl /*double*/,
                    vu /*double*/, il /*int*/, iu /*int*/, h_meig /*int **/,
                    w /*double **/, buffer /*double **/, buffer_size /*int*/,
                    info /*int **/);
  // End
}
