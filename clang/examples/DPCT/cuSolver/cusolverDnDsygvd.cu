
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, double *a,
          int lda, double *b, int ldb, double *w, double *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnDsygvd(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*double **/, lda /*int*/, b /*double **/,
                   ldb /*int*/, w /*double **/, buffer /*double **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
