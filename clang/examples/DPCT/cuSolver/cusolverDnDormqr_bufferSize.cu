
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k, const double *a,
          int lda, const double *tau, const double *c, int ldc) {
  // Start
  int buffer_size;
  hipsolverDnDormqr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      trans /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const double **/, lda /*int*/, tau /*const double **/,
      c /*const double **/, ldc /*int*/, &buffer_size /*int **/);
  // End
}
