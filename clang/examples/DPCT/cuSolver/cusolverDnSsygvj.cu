
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, float *a,
          int lda, float *b, int ldb, float *w, float *buffer, int buffer_size,
          int *info, hipsolverSyevjInfo_t params) {
  // Start
  hipsolverDnSsygvj(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*float **/, lda /*int*/, b /*float **/,
                   ldb /*int*/, w /*float **/, buffer /*float **/,
                   buffer_size /*int*/, info /*int **/, params /*syevjInfo_t*/);
  // End
}
