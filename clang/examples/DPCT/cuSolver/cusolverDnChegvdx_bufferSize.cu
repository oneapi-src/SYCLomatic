
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, const hipComplex *a, int lda,
          const hipComplex *b, int ldb, float vl, float vu, int il, int iu,
          int *h_meig, const float *w) {
  // Start
  int buffer_size;
  hipsolverDnChegvdx_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const cuComplex **/,
      lda /*int*/, b /*const cuComplex **/, ldb /*int*/, vl /*float*/,
      vu /*float*/, il /*int*/, iu /*int*/, h_meig /*int **/,
      w /*const float **/, &buffer_size /*int **/);
  // End
}
