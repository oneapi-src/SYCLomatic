#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n,
          hipDataType a_type, void *a, int64_t lda, hipDataType w_type,
          void *w, hipDataType compute_type, void *buffer, size_t buffer_size,
          int *info) {
  // Start
  cusolverDnSyevd(handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
                  jobz /*hipsolverEigMode_t*/, uplo /*hipblasFillMode_t*/,
                  n /*int64_t*/, a_type /*hipDataType*/, a /*void **/,
                  lda /*int64_t*/, w_type /*hipDataType*/, w /*void **/,
                  compute_type /*hipDataType*/, buffer /*void **/,
                  buffer_size /*size_t*/, info /*int **/);
  // End
}
