
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs,
          const double *a, int lda, double *b, int ldb, int *info) {
  // Start
  hipsolverDnDpotrs(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, nrhs /*int*/, a /*const double **/, lda /*int*/,
                   b /*double **/, ldb /*int*/, info /*int **/);
  // End
}
