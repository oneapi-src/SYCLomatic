
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *b, int ldb,
          const double *w, hipsolverSyevjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnZhegvj_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/, b /*const cuDoubleComplex **/,
      ldb /*int*/, w /*const double **/, &buffer_size /*int **/,
      params /*syevjInfo_t*/);
  // End
}
