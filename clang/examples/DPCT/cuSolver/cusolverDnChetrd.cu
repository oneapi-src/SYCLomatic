
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, float *d, float *e, hipComplex *tau, hipComplex *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnChetrd(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuComplex **/, lda /*int*/, d /*float **/,
                   e /*float **/, tau /*cuComplex **/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
