
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, hipDoubleComplex *a, int lda,
          hipDoubleComplex *b, int ldb, double vl, double vu, int il, int iu,
          int *h_meig, double *w, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZhegvdx(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, range /*cusolverEigRange_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*cuDoubleComplex **/,
      lda /*int*/, b /*cuDoubleComplex **/, ldb /*int*/, vl /*double*/,
      vu /*double*/, il /*int*/, iu /*int*/, h_meig /*int **/, w /*double **/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
