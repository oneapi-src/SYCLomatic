
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, const hipDoubleComplex *tau,
          hipDoubleComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnZungtr(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*cuDoubleComplex **/, lda /*int*/, tau /*const cuDoubleComplex **/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
