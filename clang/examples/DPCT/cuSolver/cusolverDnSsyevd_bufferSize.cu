
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const float *a, int lda,
          const float *w) {
  // Start
  int buffer_size;
  hipsolverDnSsyevd_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const float **/, lda /*int*/,
      w /*const float **/, &buffer_size /*int **/);
  // End
}
