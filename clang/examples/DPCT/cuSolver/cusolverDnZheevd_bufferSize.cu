
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const hipDoubleComplex *a, int lda,
          const double *w) {
  // Start
  int buffer_size;
  hipsolverDnZheevd_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const cuDoubleComplex **/,
      lda /*int*/, w /*const double **/, &buffer_size /*int **/);
  // End
}
