
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params, int64_t m,
          int64_t n, hipDataType a_type, const void *a, int64_t lda,
          hipDataType tau_type, const void *tau, hipDataType compute_type) {
  // Start
  size_t device_buffer_size;
  size_t host_buffer_size;
  hipsolverDnXgeqrf_bufferSize(
      handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
      m /*int64_t*/, n /*int64_t*/, a_type /*cudaDataType*/, a /*const void **/,
      lda /*int64_t*/, tau_type /*cudaDataType*/, tau /*const void **/,
      compute_type /*cudaDataType*/, &device_buffer_size /*size_t **/,
      &host_buffer_size /*size_t **/);
  // End
}
