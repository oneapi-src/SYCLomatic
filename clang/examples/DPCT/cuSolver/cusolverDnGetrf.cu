#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params, int64_t m,
          int64_t n, hipDataType a_type, void *a, int64_t lda, int64_t *ipiv,
          hipDataType compute_type, void *buffer, size_t buffer_size,
          int *info) {
  // Start
  cusolverDnGetrf(handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
                  m /*int64_t*/, n /*int64_t*/, a_type /*hipDataType*/,
                  a /*void **/, lda /*int64_t*/, ipiv /*int64_t **/,
                  compute_type /*hipDataType*/, buffer /*void **/,
                  buffer_size /*size_t*/, info /*int **/);
  // End
}
