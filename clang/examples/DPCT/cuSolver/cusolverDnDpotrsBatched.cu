
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t upper_lower, int n,
          int nrhs, double **a, int lda, double **b, int ldb, int *info,
          int group_count) {
  // Start
  hipsolverDnDpotrsBatched(
      handle /*cusolverDnHandle_t*/, upper_lower /*cublasFillMode_t*/,
      n /*int*/, nrhs /*int*/, a /*double ***/, lda /*int*/, b /*double ***/,
      ldb /*int*/, info /*int **/, group_count /*int*/);
  // End
}
