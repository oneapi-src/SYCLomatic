
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n,
          hipComplex *a, int lda, hipComplex *tau, hipComplex *c, int ldc,
          hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCunmtr(handle /*cusolverDnHandle_t*/,
                   left_right /*cublasSideMode_t*/, uplo /*cublasFillMode_t*/,
                   trans /*cublasOperation_t*/, m /*int*/, n /*int*/,
                   a /*cuComplex **/, lda /*int*/, tau /*cuComplex **/,
                   c /*cuComplex **/, ldc /*int*/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
