
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, hipComplex *a, int lda,
          const hipComplex *tau, hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCungqr(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   k /*int*/, a /*cuComplex **/, lda /*int*/,
                   tau /*const cuComplex **/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
