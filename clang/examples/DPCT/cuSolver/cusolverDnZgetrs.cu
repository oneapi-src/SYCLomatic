
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const hipDoubleComplex *a, int lda, const int *ipiv,
          hipDoubleComplex *b, int ldb, int *info) {
  // Start
  hipsolverDnZgetrs(handle /*cusolverDnHandle_t*/, trans /*cublasOperation_t*/,
                   n /*int*/, nrhs /*int*/, a /*const cuDoubleComplex **/,
                   lda /*int*/, ipiv /*const int **/, b /*cuDoubleComplex **/,
                   ldb /*int*/, info /*int **/);
  // End
}
