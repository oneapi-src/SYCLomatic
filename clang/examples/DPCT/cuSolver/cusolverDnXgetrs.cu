
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipblasOperation_t trans, int64_t n, int64_t nrhs, hipDataType a_type,
          const void *a, int64_t lda, const int64_t *ipiv, hipDataType b_type,
          void *b, int64_t ldb, int *info) {
  // Start
  hipsolverDnXgetrs(handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
                   trans /*cublasOperation_t*/, n /*int64_t*/, nrhs /*int64_t*/,
                   a_type /*cudaDataType*/, a /*const void **/, lda /*int64_t*/,
                   ipiv /*const int64_t **/, b_type /*cudaDataType*/,
                   b /*void **/, ldb /*int64_t*/, info /*int **/);
  // End
}
