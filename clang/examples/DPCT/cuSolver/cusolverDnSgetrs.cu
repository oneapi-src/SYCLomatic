
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const float *a, int lda, const int *ipiv, float *b, int ldb,
          int *info) {
  // Start
  hipsolverDnSgetrs(handle /*cusolverDnHandle_t*/, trans /*cublasOperation_t*/,
                   n /*int*/, nrhs /*int*/, a /*const float **/, lda /*int*/,
                   ipiv /*const int **/, b /*float **/, ldb /*int*/,
                   info /*int **/);
  // End
}
