
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, float *a, int lda,
          const float *tau, float *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnSorgqr(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   k /*int*/, a /*float **/, lda /*int*/, tau /*const float **/,
                   buffer /*float **/, buffer_size /*int*/, info /*int **/);
  // End
}
