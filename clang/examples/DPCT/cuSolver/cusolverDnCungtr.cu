
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, const hipComplex *tau, hipComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnCungtr(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuComplex **/, lda /*int*/,
                   tau /*const cuComplex **/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
