#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          signed char jobu, signed char jobvt, int64_t m, int64_t n,
          hipDataType a_type, void *a, int64_t lda, hipDataType s_type,
          void *s, hipDataType u_type, void *u, int64_t ldu,
          hipDataType vt_type, void *vt, int64_t ldvt,
          hipDataType compute_type, void *buffer, size_t buffer_size,
          int *info) {
  // Start
  cusolverDnGesvd(handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
                  jobu /*signed char*/, jobvt /*signed char*/, m /*int64_t*/,
                  n /*int64_t*/, a_type /*hipDataType*/, a /*void **/,
                  lda /*int64_t*/, s_type /*hipDataType*/, s /*void **/,
                  u_type /*hipDataType*/, u /*void **/, ldu /*int64_t*/,
                  vt_type /*hipDataType*/, vt /*void **/, ldvt /*int64_t*/,
                  compute_type /*hipDataType*/, buffer /*void **/,
                  buffer_size /*size_t*/, info /*int **/);
  // End
}
