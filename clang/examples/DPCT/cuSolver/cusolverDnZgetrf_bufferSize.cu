
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipDoubleComplex *a,
          int lda) {
  // Start
  int buffer_size;
  hipsolverDnZgetrf_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
                              &buffer_size /*int **/);
  // End
}
