
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          int k, double *a, int lda, const double *tau, double *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnDorgbr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/, m /*int*/,
      n /*int*/, k /*int*/, a /*double **/, lda /*int*/, tau /*const double **/,
      buffer /*double **/, buffer_size /*int*/, info /*int **/);
  // End
}
