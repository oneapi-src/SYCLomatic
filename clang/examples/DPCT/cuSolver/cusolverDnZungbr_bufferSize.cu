
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          int k, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnZungbr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/, m /*int*/,
      n /*int*/, k /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, &buffer_size /*int **/);
  // End
}
