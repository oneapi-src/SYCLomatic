
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipComplex *a, int lda) {
  // Start
  int buffer_size;
  hipsolverDnCgetrf_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, a /*cuComplex **/, lda /*int*/,
                              &buffer_size /*int **/);
  // End
}
