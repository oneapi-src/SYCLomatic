
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float *a,
          int lda, const float *tau, float *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnSorgtr(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*float **/, lda /*int*/, tau /*const float **/,
                   buffer /*float **/, buffer_size /*int*/, info /*int **/);
  // End
}
