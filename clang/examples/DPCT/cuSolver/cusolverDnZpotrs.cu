
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, int nrhs,
          const hipDoubleComplex *a, int lda, hipDoubleComplex *b, int ldb,
          int *info) {
  // Start
  hipsolverDnZpotrs(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, nrhs /*int*/, a /*const cuDoubleComplex **/,
                   lda /*int*/, b /*cuDoubleComplex **/, ldb /*int*/,
                   info /*int **/);
  // End
}
