
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int64_t n,
          hipDataType a_type, void *a, int64_t lda, hipDataType w_type,
          void *w, hipDataType compute_type, void *device_buffer,
          size_t device_buffer_size, void *host_buffer, size_t host_buffer_size,
          int *info) {
  // Start
  cusolverDnXsyevd(handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int64_t*/, a_type /*cudaDataType*/, a /* void **/,
                   lda /*int64_t*/, w_type /*cudaDataType*/, w /* void **/,
                   compute_type /*cudaDataType*/, device_buffer /*void **/,
                   device_buffer_size /*size_t*/, host_buffer /*void **/,
                   host_buffer_size /*size_t*/, info /*int **/);
  // End
}
