
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n,
          const hipComplex *a, int lda, const hipComplex *tau, const hipComplex *c,
          int ldc) {
  // Start
  int buffer_size;
  hipsolverDnCunmtr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      uplo /*cublasFillMode_t*/, trans /*cublasOperation_t*/, m /*int*/,
      n /*int*/, a /*const cuComplex **/, lda /*int*/,
      tau /*const cuComplex **/, c /*const cuComplex **/, ldc /*int*/,
      &buffer_size /*int **/);
  // End
}
