
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, const float *a,
          int lda, const float *tau) {
  // Start
  int buffer_size;
  hipsolverDnSorgqr_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, k /*int*/, a /*const float **/,
                              lda /*int*/, tau /*const float **/,
                              &buffer_size /*int **/);
  // End
}
