
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, hipDoubleComplex *a, int lda, double *s, hipDoubleComplex *u,
          int ldu, hipDoubleComplex *v, int ldv, hipDoubleComplex *buffer,
          int buffer_size, int *info, hipsolverGesvdjInfo_t params) {
  // Start
  hipsolverDnZgesvdj(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                    econ /*int*/, m /*int*/, n /*int*/, a /*cuDoubleComplex **/,
                    lda /*int*/, s /*double **/, u /*cuDoubleComplex **/,
                    ldu /*int*/, v /*cuDoubleComplex **/, ldv /*int*/,
                    buffer /*cuDoubleComplex **/, buffer_size /*int*/,
                    info /*int **/, params /*gesvdjInfo_t*/);
  // End
}
