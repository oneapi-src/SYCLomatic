#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipblasFillMode_t uplo, int64_t n, hipDataType a_type, void *a,
          int64_t lda, hipDataType compute_type, void *buffer,
          size_t buffer_size, int *info) {
  // Start
  cusolverDnPotrf(handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
                  uplo /*hipblasFillMode_t*/, n /*int64_t*/,
                  a_type /*hipDataType*/, a /*void **/, lda /*int64_t*/,
                  compute_type /*hipDataType*/, buffer /*void **/,
                  buffer_size /*size_t*/, info /*int **/);
  // End
}
