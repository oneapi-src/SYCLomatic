
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const float *a,
          int lda, const float *b, int ldb, const float *w,
          hipsolverSyevjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnSsygvj_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const float **/, lda /*int*/, b /*const float **/, ldb /*int*/,
      w /*const float **/, &buffer_size /*int **/, params /*syevjInfo_t*/);
  // End
}
