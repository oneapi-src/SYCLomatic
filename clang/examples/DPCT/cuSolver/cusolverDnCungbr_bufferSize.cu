
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          int k, const hipComplex *a, int lda, const hipComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnCungbr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/, m /*int*/,
      n /*int*/, k /*int*/, a /*const cuComplex **/, lda /*int*/,
      tau /*const cuComplex **/, &buffer_size /*int **/);
  // End
}
