
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k, const hipComplex *a,
          int lda, const hipComplex *tau, hipComplex *c, int ldc,
          hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCunmqr(handle /*cusolverDnHandle_t*/,
                   left_right /*cublasSideMode_t*/, trans /*cublasOperation_t*/,
                   m /*int*/, n /*int*/, k /*int*/, a /*const cuComplex **/,
                   lda /*int*/, tau /*const cuComplex **/, c /*cuComplex **/,
                   ldc /*int*/, buffer /*cuComplex **/, buffer_size /*int*/,
                   info /*int **/);
  // End
}
