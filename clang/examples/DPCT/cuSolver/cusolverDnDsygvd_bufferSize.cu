
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, const double *a,
          int lda, const double *b, int ldb, const double *w) {
  // Start
  int buffer_size;
  hipsolverDnDsygvd_bufferSize(
      handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
      jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const double **/, lda /*int*/, b /*const double **/, ldb /*int*/,
      w /*const double **/, &buffer_size /*int **/);
  // End
}
