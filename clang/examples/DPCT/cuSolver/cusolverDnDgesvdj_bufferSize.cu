
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, const double *a, int lda, const double *s, const double *u,
          int ldu, const double *v, int ldv, hipsolverGesvdjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnDgesvdj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/, econ /*int*/,
      m /*int*/, n /*int*/, a /*const double **/, lda /*int*/,
      s /*const double **/, u /*const double **/, ldu /*int*/,
      v /*const double **/, ldv /*int*/, &buffer_size /*int **/,
      params /*gesvdjInfo_t*/);
  // End
}
