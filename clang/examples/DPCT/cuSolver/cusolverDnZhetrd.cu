
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, double *d, double *e,
          hipDoubleComplex *tau, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZhetrd(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
                   d /*double **/, e /*double **/, tau /*cuDoubleComplex **/,
                   buffer /*cuDoubleComplex **/, buffer_size /*int*/,
                   info /*int **/);
  // End
}
