
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const double *a, int lda,
          const double *w) {
  // Start
  int buffer_size;
  hipsolverDnDsyevd_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const double **/, lda /*int*/,
      w /*const double **/, &buffer_size /*int **/);
  // End
}
