
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k, const hipComplex *a, int lda,
          const hipComplex *tau, const hipComplex *c, int ldc) {
  // Start
  int buffer_size;
  hipsolverDnCunmqr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      trans /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const cuComplex **/, lda /*int*/, tau /*const cuComplex **/,
      c /*const cuComplex **/, ldc /*int*/, &buffer_size /*int **/);
  // End
}
