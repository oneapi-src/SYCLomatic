
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, hipDoubleComplex *b, int ldb, double *w,
          hipDoubleComplex *buffer, int buffer_size, int *info,
          hipsolverSyevjInfo_t params) {
  // Start
  hipsolverDnZhegvj(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
                   b /*cuDoubleComplex **/, ldb /*int*/, w /*double **/,
                   buffer /*cuDoubleComplex **/, buffer_size /*int*/,
                   info /*int **/, params /*syevjInfo_t*/);
  // End
}
