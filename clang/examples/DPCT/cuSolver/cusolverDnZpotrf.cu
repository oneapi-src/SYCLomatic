
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZpotrf(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
                   buffer /*cuDoubleComplex **/, buffer_size /*int*/,
                   info /*int **/);
  // End
}
