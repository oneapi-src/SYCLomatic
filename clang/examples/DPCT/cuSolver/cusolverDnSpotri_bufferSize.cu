
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, float *a,
          int lda) {
  // Start
  int buffer_size;
  hipsolverDnSpotri_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*float **/, lda /*int*/, &buffer_size /*int **/);
  // End
}
