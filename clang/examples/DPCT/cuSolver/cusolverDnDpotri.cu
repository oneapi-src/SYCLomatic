
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double *a,
          int lda, double *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnDpotri(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*double **/, lda /*int*/, buffer /*double **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
