
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double *a,
          int lda, const double *tau, double *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnDorgtr(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*double **/, lda /*int*/,
                   tau /*const double **/, buffer /*double **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
