
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const double *a, int lda, const double *tau) {
  // Start
  int buffer_size;
  hipsolverDnDorgtr_bufferSize(handle /*cusolverDnHandle_t*/,
                              uplo /*cublasFillMode_t*/, n /*int*/,
                              a /*const double **/, lda /*int*/,
                              tau /*const double **/, &buffer_size /*int **/);
  // End
}
