
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipblasFillMode_t uplo, int64_t n, int64_t nrhs, hipDataType a_type,
          const void *a, int64_t lda, hipDataType b_type, void *b, int64_t ldb,
          int *info) {
  // Start
  hipsolverDnXpotrs(handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
                   uplo /*cublasFillMode_t*/, n /*int64_t*/, nrhs /*int64_t*/,
                   a_type /*cudaDataType*/, a /*const void **/, lda /*int64_t*/,
                   b_type /*cudaDataType*/, b /*void **/, ldb /*int64_t*/,
                   info /*int **/);
  // End
}
