
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const hipComplex *a, int lda, const hipComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnCungtr_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuComplex **/, lda /*int*/, tau /*const cuComplex **/,
      &buffer_size /*int **/);
  // End
}
