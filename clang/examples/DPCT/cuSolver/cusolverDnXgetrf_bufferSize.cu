
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params, int64_t m,
          int64_t n, hipDataType a_type, const void *a, int64_t lda,
          hipDataType compute_type) {
  // Start
  size_t device_buffer_size;
  size_t host_buffer_size;
  hipsolverDnXgetrf_bufferSize(
      handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
      m /*int64_t*/, n /*int64_t*/, a_type /*cudaDataType*/, a /*const void **/,
      lda /*int64_t*/, compute_type /*cudaDataType*/,
      &device_buffer_size /*size_t **/, &host_buffer_size /*size_t **/);
  // End
}
