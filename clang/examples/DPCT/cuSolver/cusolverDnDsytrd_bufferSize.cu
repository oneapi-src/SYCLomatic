
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const double *a, int lda, const double *d, const double *e,
          const double *tau) {
  // Start
  int buffer_size;
  hipsolverDnDsytrd_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const double **/, lda /*int*/, d /*const double **/,
      e /*const double **/, tau /*const double **/, &buffer_size /*int **/);
  // End
}
