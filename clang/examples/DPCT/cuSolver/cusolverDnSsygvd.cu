
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigType_t itype,
          hipsolverEigMode_t jobz, hipblasFillMode_t uplo, int n, float *a,
          int lda, float *b, int ldb, float *w, float *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnSsygvd(handle /*cusolverDnHandle_t*/, itype /*cusolverEigType_t*/,
                   jobz /*cusolverEigMode_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*float **/, lda /*int*/, b /*float **/,
                   ldb /*int*/, w /*float **/, buffer /*float **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
