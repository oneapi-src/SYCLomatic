
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t uplo, hipblasOperation_t trans, int m, int n,
          hipDoubleComplex *a, int lda, hipDoubleComplex *tau, hipDoubleComplex *c,
          int ldc, hipDoubleComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnZunmtr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      uplo /*cublasFillMode_t*/, trans /*cublasOperation_t*/, m /*int*/,
      n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
      tau /*cuDoubleComplex **/, c /*cuDoubleComplex **/, ldc /*int*/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
