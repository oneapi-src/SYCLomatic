
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, hipComplex *a,
          int lda, hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCpotri(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuComplex **/, lda /*int*/,
                   buffer /*cuComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
