
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const hipComplex *a, int lda,
          const float *w) {
  // Start
  int buffer_size;
  hipsolverDnCheevd_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const cuComplex **/,
      lda /*int*/, w /*const float **/, &buffer_size /*int **/);
  // End
}
