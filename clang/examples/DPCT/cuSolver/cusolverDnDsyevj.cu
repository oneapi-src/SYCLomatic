
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, double *a, int lda, double *w,
          double *buffer, int buffer_size, int *info, hipsolverSyevjInfo_t params) {
  // Start
  hipsolverDnDsyevj(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                   uplo /*cublasFillMode_t*/, n /*int*/, a /*double **/,
                   lda /*int*/, w /*double **/, buffer /*double **/,
                   buffer_size /*int*/, info /*int **/, params /*syevjInfo_t*/);
  // End
}
