
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          hipDoubleComplex *a, int lda, int *ipiv, hipDoubleComplex *buffer,
          int buffer_size, int *info) {
  // Start
  hipsolverDnZsytrf(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
                   ipiv /*int **/, buffer /*cuDoubleComplex **/,
                   buffer_size /*int*/, info /*int **/);
  // End
}
