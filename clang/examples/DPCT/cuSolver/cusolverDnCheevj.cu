
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, hipComplex *a, int lda, float *w,
          hipComplex *buffer, int buffer_size, int *info, hipsolverSyevjInfo_t params) {
  // Start
  hipsolverDnCheevj(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                   uplo /*cublasFillMode_t*/, n /*int*/, a /*cuComplex **/,
                   lda /*int*/, w /*float **/, buffer /*cuComplex **/,
                   buffer_size /*int*/, info /*int **/, params /*syevjInfo_t*/);
  // End
}
