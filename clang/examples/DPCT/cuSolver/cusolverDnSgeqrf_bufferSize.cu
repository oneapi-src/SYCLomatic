
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, float *a, int lda) {
  // Start
  int buffer_size;
  hipsolverDnSgeqrf_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, a /*float **/, lda /*int*/,
                              &buffer_size /*int **/);
  // End
}
