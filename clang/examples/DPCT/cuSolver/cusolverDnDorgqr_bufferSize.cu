
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, const double *a,
          int lda, const double *tau) {
  // Start
  int buffer_size;
  hipsolverDnDorgqr_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, k /*int*/, a /*const double **/,
                              lda /*int*/, tau /*const double **/,
                              &buffer_size /*int **/);
  // End
}
