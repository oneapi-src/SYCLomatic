
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const hipComplex *a, int lda, const int *ipiv, hipComplex *b, int ldb,
          int *info) {
  // Start
  hipsolverDnCgetrs(handle /*cusolverDnHandle_t*/, trans /*cublasOperation_t*/,
                   n /*int*/, nrhs /*int*/, a /*const cuComplex **/,
                   lda /*int*/, ipiv /*const int **/, b /*cuComplex **/,
                   ldb /*int*/, info /*int **/);
  // End
}
