
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const float *a, int lda, const float *d, const float *e,
          const float *tau) {
  // Start
  int buffer_size;
  hipsolverDnSsytrd_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const float **/, lda /*int*/, d /*const float **/,
      e /*const float **/, tau /*const float **/, &buffer_size /*int **/);
  // End
}
