
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnZungtr_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, &buffer_size /*int **/);
  // End
}
