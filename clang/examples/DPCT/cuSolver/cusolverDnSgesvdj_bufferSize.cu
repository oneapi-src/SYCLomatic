
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, const float *a, int lda, const float *s, const float *u,
          int ldu, const float *v, int ldv, hipsolverGesvdjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnSgesvdj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/, econ /*int*/,
      m /*int*/, n /*int*/, a /*const float **/, lda /*int*/,
      s /*const float **/, u /*const float **/, ldu /*int*/,
      v /*const float **/, ldv /*int*/, &buffer_size /*int **/,
      params /*gesvdjInfo_t*/);
  // End
}
