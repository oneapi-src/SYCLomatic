
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipComplex *a, int lda,
          hipComplex *buffer, int *ipiv, int *info) {
  // Start
  hipsolverDnCgetrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuComplex **/, lda /*int*/, buffer /*cuComplex **/,
                   ipiv /*int **/, info /*int **/);
  // End
}
