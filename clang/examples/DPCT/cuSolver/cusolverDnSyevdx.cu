#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          hipsolverEigMode_t jobz, hipsolverEigRange_t range,
          hipblasFillMode_t uplo, int n, hipDataType a_type, void *a,
          int64_t lda, void *vl, void *vu, int64_t il, int64_t iu,
          int64_t *h_meig, hipDataType w_type, void *w,
          hipDataType compute_type, void *buffer, size_t buffer_size,
          int *info) {
  // Start
  cusolverDnSyevdx(handle /*hipsolverHandle_t*/, params /*hipsolverDnParams_t*/,
                   jobz /*hipsolverEigMode_t*/, range /*hipsolverEigRange_t*/,
                   uplo /*hipblasFillMode_t*/, n /*int64_t*/,
                   a_type /*hipDataType*/, a /*void **/, lda /*int64_t*/,
                   vl /*void **/, vu /*void **/, il /*int64_t*/, iu /*int64_t*/,
                   h_meig /*int64_t **/, w_type /*hipDataType*/, w /*void **/,
                   compute_type /*hipDataType*/, buffer /*void **/,
                   buffer_size /*size_t*/, info /*int **/);
  // End
}
