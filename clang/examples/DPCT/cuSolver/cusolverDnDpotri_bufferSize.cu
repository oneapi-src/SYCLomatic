
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double *a,
          int lda) {
  // Start
  int buffer_size;
  hipsolverDnDpotri_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*double **/, lda /*int*/, &buffer_size /*int **/);
  // End
}
