
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnZungqr_bufferSize(
      handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, &buffer_size /*int **/);
  // End
}
