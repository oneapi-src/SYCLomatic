
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          int k, hipDoubleComplex *a, int lda, const hipDoubleComplex *tau,
          hipDoubleComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnZungbr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/, m /*int*/,
      n /*int*/, k /*int*/, a /*cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, buffer /*cuDoubleComplex **/,
      buffer_size /*int*/, info /*int **/);
  // End
}
