
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipComplex *a, int lda,
          hipComplex *tau, hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCgeqrf(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuComplex **/, lda /*int*/, tau /*cuComplex **/,
                   buffer /*cuComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
