
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, signed char jobu, signed char jobvt, int m,
          int n, hipComplex *a, int lda, float *s, hipComplex *u, int ldu,
          hipComplex *vt, int ldvt, hipComplex *buffer, int buffer_size,
          float *buffer_for_real, int *info) {
  // Start
  hipsolverDnCgesvd(handle /*cusolverDnHandle_t*/, jobu /*signed char*/,
                   jobvt /*signed char*/, m /*int*/, n /*int*/,
                   a /*cuComplex **/, lda /*int*/, s /*float **/,
                   u /*cuComplex **/, ldu /*int*/, vt /*cuComplex **/,
                   ldvt /*int*/, buffer /*cuComplex **/, buffer_size /*int*/,
                   buffer_for_real /*float **/, info /*int **/);
  // End
}
