
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t upper_lower, int n,
          hipDoubleComplex **a, int lda, int *info, int group_count) {
  // Start
  hipsolverDnZpotrfBatched(handle /*cusolverDnHandle_t*/,
                          upper_lower /*cublasFillMode_t*/, n /*int*/,
                          a /*cuDoubleComplex ***/, lda /*int*/, info /*int **/,
                          group_count /*int*/);
  // End
}
