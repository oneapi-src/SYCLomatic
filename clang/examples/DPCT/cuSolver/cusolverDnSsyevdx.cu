
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipsolverEigRange_t range, hipblasFillMode_t uplo, int n, float *a,
          int lda, float vl, float vu, int il, int iu, int *h_meig, float *w,
          float *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnSsyevdx(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                    range /*cusolverEigRange_t*/, uplo /*cublasFillMode_t*/,
                    n /*int*/, a /*float **/, lda /*int*/, vl /*float*/,
                    vu /*float*/, il /*int*/, iu /*int*/, h_meig /*int **/,
                    w /*float **/, buffer /*float **/, buffer_size /*int*/,
                    info /*int **/);
  // End
}
