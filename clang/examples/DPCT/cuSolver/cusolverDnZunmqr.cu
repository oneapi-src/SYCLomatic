
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *tau,
          hipDoubleComplex *c, int ldc, hipDoubleComplex *buffer, int buffer_size,
          int *info) {
  // Start
  hipsolverDnZunmqr(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      trans /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, c /*cuDoubleComplex **/, ldc /*int*/,
      buffer /*cuDoubleComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
