
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, const hipDoubleComplex *a, int lda,
          const double *w, hipsolverSyevjInfo_t params) {
  // Start
  int buffer_size;
  hipsolverDnZheevj_bufferSize(
      handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
      uplo /*cublasFillMode_t*/, n /*int*/, a /*const cuDoubleComplex **/,
      lda /*int*/, w /*const double **/, &buffer_size /*int **/,
      params /*syevjInfo_t*/);
  // End
}
