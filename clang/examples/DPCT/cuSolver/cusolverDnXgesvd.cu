
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverDnParams_t params,
          signed char jobu, signed char jobvt, int64_t m, int64_t n,
          hipDataType a_type, void *a, int64_t lda, hipDataType s_type,
          void *s, hipDataType u_type, void *u, int64_t ldu,
          hipDataType vt_type, void *vt, int64_t ldvt,
          hipDataType compute_type, void *device_buffer,
          size_t device_buffer_size, void *host_buffer, size_t host_buffer_size,
          int *info) {
  // Start
  cusolverDnXgesvd(handle /*cusolverDnHandle_t*/, params /*cusolverDnParams_t*/,
                   jobu /*signed char*/, jobvt /*signed char*/, m /*int64_t*/,
                   n /*int64_t*/, a_type /*cudaDataType*/, a /* void **/,
                   lda /*int64_t*/, s_type /*cudaDataType*/, s /* void **/,
                   u_type /*cudaDataType*/, u /* void **/, ldu /*int64_t*/,
                   vt_type /*cudaDataType*/, vt /* void **/, ldvt /*int64_t*/,
                   compute_type /*cudaDataType*/, device_buffer /*void **/,
                   device_buffer_size /*size_t*/, host_buffer /*void **/,
                   host_buffer_size /*size_t*/, info /*int **/);
  // End
}
