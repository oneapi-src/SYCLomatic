
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, hipComplex *a, int lda,
          float *d, float *e, hipComplex *tau_q, hipComplex *tau_p,
          hipComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnCgebrd(handle /*cusolverDnHandle_t*/, m /*int*/, n /*int*/,
                   a /*cuComplex **/, lda /*int*/, d /*float **/, e /*float **/,
                   tau_q /*cuComplex **/, tau_p /*cuComplex **/,
                   buffer /*cuComplex **/, buffer_size /*int*/, info /*int **/);
  // End
}
