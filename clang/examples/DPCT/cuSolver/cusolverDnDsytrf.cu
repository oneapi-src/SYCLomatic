
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n, double *a,
          int lda, int *ipiv, double *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnDsytrf(handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/,
                   n /*int*/, a /*double **/, lda /*int*/, ipiv /*int **/,
                   buffer /*double **/, buffer_size /*int*/, info /*int **/);
  // End
}
