
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz,
          hipblasFillMode_t uplo, int n, hipDoubleComplex *a, int lda, double *w,
          hipDoubleComplex *buffer, int buffer_size, int *info) {
  // Start
  hipsolverDnZheevd(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                   uplo /*cublasFillMode_t*/, n /*int*/,
                   a /*cuDoubleComplex **/, lda /*int*/, w /*double **/,
                   buffer /*cuDoubleComplex **/, buffer_size /*int*/,
                   info /*int **/);
  // End
}
