
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t upper_lower, int n,
          int nrhs, hipComplex **a, int lda, hipComplex **b, int ldb, int *info,
          int group_count) {
  // Start
  hipsolverDnCpotrsBatched(
      handle /*cusolverDnHandle_t*/, upper_lower /*cublasFillMode_t*/,
      n /*int*/, nrhs /*int*/, a /*cuComplex ***/, lda /*int*/,
      b /*cuComplex ***/, ldb /*int*/, info /*int **/, group_count /*int*/);
  // End
}
