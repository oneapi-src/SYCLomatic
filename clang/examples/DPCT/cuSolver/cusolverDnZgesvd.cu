
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, signed char jobu, signed char jobvt, int m,
          int n, hipDoubleComplex *a, int lda, double *s, hipDoubleComplex *u,
          int ldu, hipDoubleComplex *vt, int ldvt, hipDoubleComplex *buffer,
          int buffer_size, double *buffer_for_real, int *info) {
  // Start
  hipsolverDnZgesvd(
      handle /*cusolverDnHandle_t*/, jobu /*signed char*/,
      jobvt /*signed char*/, m /*int*/, n /*int*/, a /*cuDoubleComplex **/,
      lda /*int*/, s /*double **/, u /*cuDoubleComplex **/, ldu /*int*/,
      vt /*cuDoubleComplex **/, ldvt /*int*/, buffer /*cuDoubleComplex **/,
      buffer_size /*int*/, buffer_for_real /*double **/, info /*int **/);
  // End
}
