
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasFillMode_t uplo, int n,
          const hipComplex *a, int lda, const float *d, const float *e,
          const hipComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnChetrd_bufferSize(
      handle /*cusolverDnHandle_t*/, uplo /*cublasFillMode_t*/, n /*int*/,
      a /*const cuComplex **/, lda /*int*/, d /*const float **/,
      e /*const float **/, tau /*const cuComplex **/, &buffer_size /*int **/);
  // End
}
