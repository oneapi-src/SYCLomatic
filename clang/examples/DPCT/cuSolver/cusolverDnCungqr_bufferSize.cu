
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, int m, int n, int k, const hipComplex *a,
          int lda, const hipComplex *tau) {
  // Start
  int buffer_size;
  hipsolverDnCungqr_bufferSize(handle /*cusolverDnHandle_t*/, m /*int*/,
                              n /*int*/, k /*int*/, a /*const cuComplex **/,
                              lda /*int*/, tau /*const cuComplex **/,
                              &buffer_size /*int **/);
  // End
}
