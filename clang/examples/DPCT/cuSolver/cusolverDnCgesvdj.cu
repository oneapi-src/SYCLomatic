
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipsolverEigMode_t jobz, int econ, int m,
          int n, hipComplex *a, int lda, float *s, hipComplex *u, int ldu,
          hipComplex *v, int ldv, hipComplex *buffer, int buffer_size, int *info,
          hipsolverGesvdjInfo_t params) {
  // Start
  hipsolverDnCgesvdj(handle /*cusolverDnHandle_t*/, jobz /*cusolverEigMode_t*/,
                    econ /*int*/, m /*int*/, n /*int*/, a /*cuComplex **/,
                    lda /*int*/, s /*float **/, u /*cuComplex **/, ldu /*int*/,
                    v /*cuComplex **/, ldv /*int*/, buffer /*cuComplex **/,
                    buffer_size /*int*/, info /*int **/,
                    params /*gesvdjInfo_t*/);
  // End
}
