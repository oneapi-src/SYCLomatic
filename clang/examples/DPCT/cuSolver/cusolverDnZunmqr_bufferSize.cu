
#include <hip/hip_runtime.h>
#include "hipsolver.h"

void test(hipsolverHandle_t handle, hipblasSideMode_t left_right,
          hipblasOperation_t trans, int m, int n, int k,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *tau,
          const hipDoubleComplex *c, int ldc) {
  // Start
  int buffer_size;
  hipsolverDnZunmqr_bufferSize(
      handle /*cusolverDnHandle_t*/, left_right /*cublasSideMode_t*/,
      trans /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      a /*const cuDoubleComplex **/, lda /*int*/,
      tau /*const cuDoubleComplex **/, c /*const cuDoubleComplex **/,
      ldc /*int*/, &buffer_size /*int **/);
  // End
}
