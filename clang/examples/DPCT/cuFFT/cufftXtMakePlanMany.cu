
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"
#include "hipfft/hipfftXt.h"

void test(hipfftHandle plan, int dim, long long int *n, long long int *inembed,
          long long int istride, long long int idist, hipDataType itype,
          long long int *onembed, long long int ostride, long long int odist,
          hipDataType otype, long long int num_of_trans, size_t *worksize,
          hipDataType exectype) {
  // Start
  hipfftXtMakePlanMany(plan /*cufftHandle*/, dim /*int*/, n /*long long int **/,
                      inembed /*long long int **/, istride /*long long int*/,
                      idist /*long long int*/, itype /*cudaDataType*/,
                      onembed /*long long int **/, ostride /*long long int*/,
                      odist /*long long int*/, otype /*cudaDataType*/,
                      num_of_trans /*long long int*/, worksize /*size_t **/,
                      exectype /*cudaDataType*/);
  // End
}
