
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(int nx, int ny, int nz, hipfftType type) {
  // Start
  hipfftHandle plan;
  hipfftPlan3d(&plan /*cufftHandle **/, nx /*int*/, ny /*int*/, nz /*int*/,
              type /*cufftType*/);
  // End
}
