
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftReal *in, hipfftComplex *out) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecR2C(plan /*cufftHandle*/, in /*cufftReal **/,
               out /*cufftComplex **/);
  // End
}
