
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(int dim, int *n, int *inembed, int istride, int idist, int *onembed,
          int ostride, int odist, hipfftType type, int num_of_trans) {
  // Start
  hipfftHandle plan;
  hipfftPlanMany(&plan /*cufftHandle **/, dim /*int*/, n /*int **/,
                inembed /*int **/, istride /*int*/, idist /*int*/,
                onembed /*int **/, ostride /*int*/, odist /*int*/,
                type /*cufftType*/, num_of_trans /*int*/);
  // End
}
