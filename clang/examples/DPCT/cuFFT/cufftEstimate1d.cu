
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, hipfftType type, int num_of_trans,
          size_t *worksize) {
  // Start
  hipfftEstimate1d(nx /*int*/, type /*cufftType*/, num_of_trans /*int*/,
                  worksize /*size_t **/);
  // End
}
