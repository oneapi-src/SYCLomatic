
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, hipfftType type, int num_of_trans,
          size_t *worksize) {
  // Start
  hipfftGetSize1d(plan /*cufftHandle*/, nx /*int*/, type /*cufftType*/,
                 num_of_trans /*int*/, worksize /*size_t **/);
  // End
}
