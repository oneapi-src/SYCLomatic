
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftComplex *in, hipfftReal *out) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecC2R(plan /*cufftHandle*/, in /*cufftComplex **/,
               out /*cufftReal **/);
  // End
}
