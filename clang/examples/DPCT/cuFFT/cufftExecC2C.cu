
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftComplex *in, hipfftComplex *out, int dir) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecC2C(plan /*cufftHandle*/, in /*cufftComplex **/,
               out /*cufftComplex **/, dir /*int*/);
  // End
}
