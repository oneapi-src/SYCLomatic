
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, hipfftDoubleReal *in, hipfftDoubleComplex *out) {
  // Start
  hipfftExecD2Z(plan /*cufftHandle*/, in /*cufftDoubleReal **/,
               out /*cufftDoubleComplex **/);
  // End
}
