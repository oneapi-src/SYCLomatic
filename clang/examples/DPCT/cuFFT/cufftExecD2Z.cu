
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftDoubleReal *in, hipfftDoubleComplex *out) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecD2Z(plan /*cufftHandle*/, in /*cufftDoubleReal **/,
               out /*cufftDoubleComplex **/);
  // End
}
