
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, int ny, hipfftType type, size_t *worksize) {
  // Start
  hipfftMakePlan2d(plan /*cufftHandle*/, nx /*int*/, ny /*int*/,
                  type /*cufftType*/, worksize /*size_t **/);
  // End
}
