
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, hipfftDoubleComplex *in, hipfftDoubleComplex *out,
          int dir) {
  // Start
  hipfftExecZ2Z(plan /*cufftHandle*/, in /*cufftDoubleComplex **/,
               out /*cufftDoubleComplex **/, dir /*int*/);
  // End
}
