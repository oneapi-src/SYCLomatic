
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftDoubleComplex *in, hipfftDoubleComplex *out, int dir) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecZ2Z(plan /*cufftHandle*/, in /*cufftDoubleComplex **/,
               out /*cufftDoubleComplex **/, dir /*int*/);
  // End
}
