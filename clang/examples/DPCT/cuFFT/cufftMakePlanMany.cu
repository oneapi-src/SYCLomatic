
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int dim, int *n, int *inembed, int istride,
          int idist, int *onembed, int ostride, int odist, hipfftType type,
          int num_of_trans, size_t *worksize) {
  // Start
  hipfftMakePlanMany(plan /*cufftHandle*/, dim /*int*/, n /*int **/,
                    inembed /*int **/, istride /*int*/, idist /*int*/,
                    onembed /*int **/, ostride /*int*/, odist /*int*/,
                    type /*cufftType*/, num_of_trans /*int*/,
                    worksize /*size_t **/);
  // End
}
