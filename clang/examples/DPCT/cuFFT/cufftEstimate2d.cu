
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, int ny, hipfftType type, size_t *worksize) {
  // Start
  hipfftEstimate2d(nx /*int*/, ny /*int*/, type /*cufftType*/,
                  worksize /*size_t **/);
  // End
}
