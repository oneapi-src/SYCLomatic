
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, int ny, int nz, hipfftType type,
          size_t *worksize) {
  // Start
  hipfftMakePlan3d(plan /*cufftHandle*/, nx /*int*/, ny /*int*/, nz /*int*/,
                  type /*cufftType*/, worksize /*size_t **/);
  // End
}
