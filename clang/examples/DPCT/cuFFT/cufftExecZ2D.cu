
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftDoubleComplex *in, hipfftDoubleReal *out) {
  // Start
  hipfftHandle plan;
  hipfftCreate(&plan /*cufftHandle **/);
  hipfftExecZ2D(plan /*cufftHandle*/, in /*cufftDoubleComplex **/,
               out /*cufftDoubleReal **/);
  // End
}
