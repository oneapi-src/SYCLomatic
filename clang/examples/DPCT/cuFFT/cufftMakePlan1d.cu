
#include <hip/hip_runtime.h>
#include "hipfft/hipfft.h"

void test(hipfftHandle plan, int nx, hipfftType type, int num_of_trans,
          size_t *worksize) {
  // Start
  hipfftMakePlan1d(plan /*cufftHandle*/, nx /*int*/, type /*cufftType*/,
                  num_of_trans /*int*/, worksize /*size_t **/);
  // End
}
