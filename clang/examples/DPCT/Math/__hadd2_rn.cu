// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h1, __half2 h2, __hip_bfloat162 b1,
                     __hip_bfloat162 b2) {
  // Start
  __hadd2_rn(h1 /*__half2*/, h2 /*__half2*/);
  __hadd2_rn(b1 /*__nv_bfloat162*/, b2 /*__nv_bfloat162*/);
  // End
}
