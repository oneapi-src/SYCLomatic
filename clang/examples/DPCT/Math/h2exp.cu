// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h, __hip_bfloat162 b) {
  // Start
  h2exp(h /*__half2*/);
  h2exp(b /*__nv_bfloat162*/);
  // End
}
