// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h1, __half h2, __hip_bfloat16 b1, __hip_bfloat16 b2) {
  // Start
  __hmin(h1 /*__half*/, h2 /*__half*/);
  __hmin(b1 /*__nv_bfloat16*/, b2 /*__nv_bfloat16*/);
  // End
}
