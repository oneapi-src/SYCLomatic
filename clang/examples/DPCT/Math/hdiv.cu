#include "hip/hip_runtime.h"
// Option: --use-dpcpp-extensions=intel_device_math
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h1, __half h2) {
  // Start
  hdiv(h1 /*__half*/, h2 /*__half*/);
  // End
}
