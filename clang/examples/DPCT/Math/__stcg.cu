
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *ph, __half h, __half2 *ph2, __half2 h2,
                     __hip_bfloat16 *pb, __hip_bfloat16 b, __hip_bfloat162 *pb2,
                     __hip_bfloat162 b2) {
  // Start
  __stcg(ph /*__half **/, h /*__half*/);
  __stcg(ph2 /*__half2 **/, h2 /*__half2*/);
  __stcg(pb /*__nv_bfloat16 **/, b /*__nv_bfloat16*/);
  __stcg(pb2 /*__nv_bfloat162 **/, b2 /*__nv_bfloat162*/);
  // End
}
