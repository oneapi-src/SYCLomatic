
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *ph, __half h, __half2 *ph2, __half2 h2,
                     __hip_bfloat16 *pb, __hip_bfloat16 b, __hip_bfloat162 *pb2,
                     __hip_bfloat162 b2) {
  // Start
  /* 1 */ __stcg(ph /*__half **/, h /*__half*/);
  /* 2 */ __stcg(ph2 /*__half2 **/, h2 /*__half2*/);
  /* 3 */ __stcg(pb /*__nv_bfloat16 **/, b /*__nv_bfloat16*/);
  /* 4 */ __stcg(pb2 /*__nv_bfloat162 **/, b2 /*__nv_bfloat162*/);
  // End
}
