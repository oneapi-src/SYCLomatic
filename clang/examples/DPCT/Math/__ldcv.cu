
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, __hip_bfloat16 *b,
                     __hip_bfloat162 *b2) {
  // Start
  /* 1 */ __ldcv(h /*__half **/);
  /* 2 */ __ldcv(h2 /*__half2 **/);
  /* 3 */ __ldcv(b /*__nv_bfloat16 **/);
  /* 4 */ __ldcv(b2 /*__nv_bfloat162 **/);
  // End
}
