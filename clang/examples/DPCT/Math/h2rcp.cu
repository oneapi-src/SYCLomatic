// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h, __hip_bfloat162 b) {
  // Start
  h2rcp(h /*__half2*/);
  h2rcp(b /*__nv_bfloat162*/);
  // End
}
