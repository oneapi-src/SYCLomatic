// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h, __hip_bfloat16 b) {
  // Start
  hexp10(h /*__half*/);
  hexp10(b /*__nv_bfloat16*/);
  // End
}
