// Option: --use-dpcpp-extensions=intel_device_math
// Option: --use-experimental-features=bfloat16_math_functions

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h, __hip_bfloat16 b) {
  // Start
  hcos(h /*__half*/);
  hcos(b /*__nv_bfloat16*/);
  // End
}
