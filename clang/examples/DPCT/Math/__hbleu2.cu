// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half2 h1, __half2 h2, __hip_bfloat162 b1,
                     __hip_bfloat162 b2) {
  // Start
  __hbleu2(h1 /*__half2*/, h2 /*__half2*/);
  __hbleu2(b1 /*__nv_bfloat162*/, b2 /*__nv_bfloat162*/);
  // End
}
