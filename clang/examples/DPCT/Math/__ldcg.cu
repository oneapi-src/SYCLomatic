
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, __hip_bfloat16 *b,
                     __hip_bfloat162 *b2) {
  // Start
  /* 1 */ __ldcg(h /*__half **/);
  /* 2 */ __ldcg(h2 /*__half2 **/);
  /* 3 */ __ldcg(b /*__nv_bfloat16 **/);
  /* 4 */ __ldcg(b2 /*__nv_bfloat162 **/);
  // End
}
