
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, __hip_bfloat16 *b,
                     __hip_bfloat162 *b2) {
  // Start
  __ldcg(h /*__half **/);
  __ldcg(h2 /*__half2 **/);
  __ldcg(b /*__nv_bfloat16 **/);
  __ldcg(b2 /*__nv_bfloat162 **/);
  // End
}
