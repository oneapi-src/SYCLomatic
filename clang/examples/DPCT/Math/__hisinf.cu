// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h, __hip_bfloat16 b) {
  // Start
  __hisinf(h /*__half*/);
  __hisinf(b /*__nv_bfloat16*/);
  // End
}
