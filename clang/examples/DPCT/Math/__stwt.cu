
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *ph, __half h, __half2 *ph2, __half2 h2,
                     __hip_bfloat16 *pb, __hip_bfloat16 b, __hip_bfloat162 *pb2,
                     __hip_bfloat162 b2) {
  // Start
  __stwt(ph /*__half **/, h /*__half*/);
  __stwt(ph2 /*__half2 **/, h2 /*__half2*/);
  __stwt(pb /*__nv_bfloat16 **/, b /*__nv_bfloat16*/);
  __stwt(pb2 /*__nv_bfloat162 **/, b2 /*__nv_bfloat162*/);
  // End
}
