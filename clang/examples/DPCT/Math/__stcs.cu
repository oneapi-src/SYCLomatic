
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *ph, __half h, __half2 *ph2, __half2 h2,
                     __hip_bfloat16 *pb, __hip_bfloat16 b, __hip_bfloat162 *pb2,
                     __hip_bfloat162 b2) {
  // Start
  __stcs(ph /*__half **/, h /*__half*/);
  __stcs(ph2 /*__half2 **/, h2 /*__half2*/);
  __stcs(pb /*__nv_bfloat16 **/, b /*__nv_bfloat16*/);
  __stcs(pb2 /*__nv_bfloat162 **/, b2 /*__nv_bfloat162*/);
  // End
}
