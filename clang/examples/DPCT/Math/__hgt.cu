// Option: --use-dpcpp-extensions=intel_device_math

#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half h1, __half h2, __hip_bfloat16 b1, __hip_bfloat16 b2) {
  // Start
  __hgt(h1 /*__half*/, h2 /*__half*/);
  __hgt(b1 /*__nv_bfloat16*/, b2 /*__nv_bfloat16*/);
  // End
}
