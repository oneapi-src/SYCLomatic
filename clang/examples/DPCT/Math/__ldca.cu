
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, __hip_bfloat16 *b,
                     __hip_bfloat162 *b2) {
  // Start
  __ldca(h /*__half **/);
  __ldca(h2 /*__half2 **/);
  __ldca(b /*__nv_bfloat16 **/);
  __ldca(b2 /*__nv_bfloat162 **/);
  // End
}
