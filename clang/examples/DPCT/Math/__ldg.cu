
#include <hip/hip_runtime.h>
#include "hip/hip_bf16.h"
#include "hip/hip_fp16.h"

__global__ void test(__half *h, __half2 *h2, __hip_bfloat16 *b,
                     __hip_bfloat162 *b2) {
  // Start
  __ldg(h /*__half **/);
  __ldg(h2 /*__half2 **/);
  __ldg(b /*__nv_bfloat16 **/);
  __ldg(b2 /*__nv_bfloat162 **/);
  // End
}
