
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          const double *a, int lda, const double *x, int incx, double *c,
          int ldc) {
  // Start
  hipblasDdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              m /*int*/, n /*int*/, a /*const double **/, lda /*int*/,
              x /*const double **/, incx /*int*/, c /*double **/, ldc /*int*/);
  // End
}
