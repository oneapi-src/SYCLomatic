
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int m, int n, const double *alpha,
          const double *a, int lda, const double *b, int ldb,
          const double *beta, double *c, int ldc) {
  // Start
  hipblasDsymm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, m /*int*/, n /*int*/,
              alpha /*const double **/, a /*const double **/, lda /*int*/,
              b /*const double **/, ldb /*int*/, beta /*const double **/,
              c /*double **/, ldc /*int*/);
  // End
}
