
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int m, int n, const float *alpha,
          const float *a, int lda, const float *b, int ldb, const float *beta,
          float *c, int ldc) {
  // Start
  hipblasSsymm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, m /*int*/, n /*int*/,
              alpha /*const float **/, a /*const float **/, lda /*int*/,
              b /*const float **/, ldb /*int*/, beta /*const float **/,
              c /*float **/, ldc /*int*/);
  // End
}
