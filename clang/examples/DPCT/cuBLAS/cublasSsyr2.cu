
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const float *alpha, const float *x, int incx, const float *y,
          int incy, float *a, int lda) {
  // Start
  hipblasSsyr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const float **/, x /*const float **/,
              incx /*int*/, y /*const float **/, incy /*int*/, a /*float **/,
              lda /*int*/);
  // End
}
