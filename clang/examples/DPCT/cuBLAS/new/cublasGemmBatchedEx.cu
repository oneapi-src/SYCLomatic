
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const void *alpha,
          const void *const *a, hipDataType atype, int lda,
          const void *const *b, hipDataType btype, int ldb, const void *beta,
          void *const *c, hipDataType ctype, int ldc, int group_count,
          hipblasComputeType_t computetype_computeType_t,
          hipDataType computetype_dataType, hipblasGemmAlgo_t algo) {
  // Start
  hipblasGemmBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void *const **/,
      atype /*cudaDataType_t*/, lda /*int*/, b /*const void *const **/,
      btype /*cudaDataType_t*/, ldb /*int*/, beta /*const void **/,
      c /*void *const **/, ctype /*cudaDataType_t*/, ldc /*int*/,
      group_count /*int*/, computetype_computeType_t /*cublasComputeType_t*/,
      algo /*cublasGemmAlgo_t*/);
  hipblasGemmBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void *const **/,
      atype /*cudaDataType_t*/, lda /*int*/, b /*const void *const **/,
      btype /*cudaDataType_t*/, ldb /*int*/, beta /*const void **/,
      c /*void *const **/, ctype /*cudaDataType_t*/, ldc /*int*/,
      group_count /*int*/, computetype_dataType /*cudaDataType*/,
      algo /*cublasGemmAlgo_t*/);
  // End
}
