
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const void *alpha,
          const void *a, hipDataType atype, int lda, long long int stridea,
          const void *b, hipDataType btype, int ldb, long long int strideb,
          const void *beta, void *c, hipDataType ctype, int ldc,
          long long int stridec, int group_count,
          hipblasComputeType_t computetype_computeType_t,
          hipDataType computetype_dataType, hipblasGemmAlgo_t algo) {
  // Start
  hipblasGemmStridedBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void **/, atype /*cudaDataType_t*/,
      lda /*int*/, stridea /*long long int*/, b /*const void **/,
      btype /*cudaDataType_t*/, ldb /*int*/, strideb /*long long int*/,
      beta /*const void **/, c /*const void **/, ctype /*cudaDataType_t*/,
      ldc /*int*/, stridec /*long long int*/, group_count /*int*/,
      computetype_computeType_t /*cublasComputeType_t*/,
      algo /*cublasGemmAlgo_t*/);
  hipblasGemmStridedBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void **/, atype /*cudaDataType_t*/,
      lda /*int*/, stridea /*long long int*/, b /*const void **/,
      btype /*cudaDataType_t*/, ldb /*int*/, strideb /*long long int*/,
      beta /*const void **/, c /*const void **/, ctype /*cudaDataType_t*/,
      ldc /*int*/, stridec /*long long int*/, group_count /*int*/,
      computetype_dataType /*cudaDataType*/, algo /*cublasGemmAlgo_t*/);
  // End
}
