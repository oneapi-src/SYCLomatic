
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const float *alpha,
          const void *a, hipDataType atype, int lda, const void *b,
          hipDataType btype, int ldb, const float *beta, void *c,
          hipDataType ctype, int ldc) {
  // Start
  cublasSgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
                alpha /*const float **/, a /*const void **/,
                atype /*cudaDataType_t*/, lda /*int*/, b /*const void **/,
                btype /*cudaDataType_t*/, ldb /*int*/, beta /*const float **/,
                c /*const void **/, ctype /*cudaDataType_t*/, ldc /*int*/);
  // End
}
