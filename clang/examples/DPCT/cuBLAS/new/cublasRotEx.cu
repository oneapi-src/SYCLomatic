
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, void *x, hipDataType xtype, int incx,
          void *y, hipDataType ytype, int incy, const void *c, const void *s,
          hipDataType cstype, hipDataType computetype) {
  // Start
  hipblasRotEx(handle /*cublasHandle_t*/, n /*int*/, x /*const void **/,
              xtype /*cudaDataType_t*/, incx /*int*/, y /*const void **/,
              ytype /*cudaDataType*/, incy /*int*/, c /*const void **/,
              s /*const void **/, cstype /*cudaDataType_t*/,
              computetype /*cudaDataType*/);
  // End
}
