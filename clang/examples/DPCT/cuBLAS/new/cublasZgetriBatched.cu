
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *const *a,
          int lda, int *ipiv, hipDoubleComplex *const *c, int ldc, int *info,
          int group_count) {
  // Start
  hipblasZgetriBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*const cuDoubleComplex *const **/, lda /*int*/,
                      ipiv /*int **/, c /*cuDoubleComplex *const **/,
                      ldc /*int*/, info /*int **/, group_count /*int*/);
  // End
}
