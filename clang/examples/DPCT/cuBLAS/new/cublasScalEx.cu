
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const void *alpha,
          hipDataType alphatype, void *x, hipDataType xtype, int incx,
          hipDataType computetype) {
  // Start
  hipblasScalEx(handle /*cublasHandle_t*/, n /*int*/, alpha /*const void **/,
               alphatype /*cudaDataType_t*/, x /*const void **/,
               xtype /*cudaDataType*/, incx /*int*/,
               computetype /*cudaDataType*/);
  // End
}
