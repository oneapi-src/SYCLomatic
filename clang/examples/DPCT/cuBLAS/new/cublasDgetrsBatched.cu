
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const double *const *a, int lda, const int *ipiv, double *const *b,
          int ldb, int *info, int group_count) {
  // Start
  hipblasDgetrsBatched(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/,
                      n /*int*/, nrhs /*int*/, a /*const double *const **/,
                      lda /*int*/, ipiv /*const int **/, b /*double *const **/,
                      ldb /*int*/, info /*int **/, group_count /*int*/);
  // End
}
