
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *beta, hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZsyrk_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
                 ldc /*int64_t*/);
  // End
}
