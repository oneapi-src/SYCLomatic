
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          long long int stridea, const hipDoubleComplex *b, int ldb,
          long long int strideb, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int ldc, long long int stridec, int group_count) {
  // Start
  hipblasZgemmStridedBatched(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
      lda /*int*/, stridea /*long long int*/, b /*const cuDoubleComplex **/,
      ldb /*int*/, strideb /*long long int*/, beta /*const cuDoubleComplex **/,
      c /*cuDoubleComplex **/, ldc /*int*/, stridec /*long long int*/,
      group_count /*int*/);
  // End
}
