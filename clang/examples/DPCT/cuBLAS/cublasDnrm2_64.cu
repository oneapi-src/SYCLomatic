
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *x, int64_t incx,
          double *res) {
  // Start
  hipblasDnrm2_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*const double **/,
                 incx /*int64_t*/, res /*double **/);
  // End
}
