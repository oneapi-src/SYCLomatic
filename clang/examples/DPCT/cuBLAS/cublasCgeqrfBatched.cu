
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int m, int n, hipComplex *const *a, int lda,
          hipComplex *const *tau, int *info, int group_count) {
  // Start
  hipblasCgeqrfBatched(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
                      a /*cuComplex *const **/, lda /*int*/,
                      tau /*cuComplex *const **/, info /*int **/,
                      group_count /*int*/);
  // End
}
