
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const double *alpha,
          const double *a, int lda, const double *b, int ldb,
          const double *beta, double *c, int ldc) {
  // Start
  hipblasDgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
              transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
              alpha /*const double **/, a /*const double **/, lda /*int*/,
              b /*const double **/, ldb /*int*/, beta /*const double **/,
              c /*double **/, ldc /*int*/);
  // End
}
