
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipDoubleComplex *const *a, int lda,
          int *ipiv, int *info, int group_count) {
  // Start
  hipblasZgetrfBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*cuDoubleComplex *const **/, lda /*int*/,
                      ipiv /*int **/, info /*int **/, group_count /*int*/);
  // End
}
