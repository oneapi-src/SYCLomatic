
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *x, int incx,
          hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              m /*int*/, n /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
