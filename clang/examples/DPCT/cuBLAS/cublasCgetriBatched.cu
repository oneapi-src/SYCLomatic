
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *const *a, int lda,
          const int *ipiv, hipComplex *const *c, int ldc, int *info,
          int group_count) {
  // Start
  hipblasCgetriBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*const cuComplex *const **/, lda /*int*/,
                      ipiv /*const int **/, c /*cuComplex *const **/,
                      ldc /*int*/, info /*int **/, group_count /*int*/);
  // End
}
