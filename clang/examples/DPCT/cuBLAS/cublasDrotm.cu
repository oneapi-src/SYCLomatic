
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, double *x, int incx, double *y,
          int incy, const double *param) {
  // Start
  hipblasDrotm(handle /*cublasHandle_t*/, n /*int*/, x /*double **/,
              incx /*int*/, y /*double **/, incy /*int*/,
              param /*const double **/);
  // End
}
