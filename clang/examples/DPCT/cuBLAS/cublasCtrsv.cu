
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n,
          const hipComplex *a, int lda, hipComplex *x, int incx) {
  // Start
  hipblasCtrsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, a /*const cuComplex **/, lda /*int*/,
              x /*cuComplex **/, incx /*int*/);
  // End
}
