
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const double *alpha,
          const hipDoubleComplex *a, int lda, const double *beta,
          hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZherk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
              alpha /*const double **/, a /*const cuDoubleComplex **/,
              lda /*int*/, beta /*const double **/, c /*cuDoubleComplex **/,
              ldc /*int*/);
  // End
}
