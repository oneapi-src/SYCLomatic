
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const float *alpha,
          const float *a, int lda, const float *b, int ldb, const float *beta,
          float *c, int ldc) {
  // Start
  hipblasSsyrkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const float **/, a /*const float **/, lda /*int*/,
               b /*const float **/, ldb /*int*/, beta /*const float **/,
               c /*float **/, ldc /*int*/);
  // End
}
