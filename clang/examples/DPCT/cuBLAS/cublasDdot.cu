
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *x, int incx,
          const double *y, int incy, double *res) {
  // Start
  hipblasDdot(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
             incx /*int*/, y /*const double **/, incy /*int*/,
             res /*double **/);
  // End
}
