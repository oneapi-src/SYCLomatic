
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const double *alpha,
          const double *const *a, int lda, double *const *b, int ldb,
          int group_count) {
  // Start
  hipblasDtrsmBatched(
      handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
      upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
      unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
      alpha /*const double **/, a /*const double *const **/, lda /*int*/,
      b /*double *const **/, ldb /*int*/, group_count /*int*/);
  // End
}
