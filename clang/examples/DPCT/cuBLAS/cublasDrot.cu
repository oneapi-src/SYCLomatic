
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, double *x, int incx, double *y,
          int incy, const double *c, const double *s) {
  // Start
  hipblasDrot(handle /*cublasHandle_t*/, n /*int*/, x /*double **/, incx /*int*/,
             y /*double **/, incy /*int*/, c /*const double **/,
             s /*const double **/);
  // End
}
