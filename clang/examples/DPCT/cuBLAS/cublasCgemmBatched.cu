
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const hipComplex *alpha,
          const hipComplex *const *a, int lda, const hipComplex *const *b,
          int ldb, const hipComplex *beta, hipComplex *const *c, int ldc,
          int group_count) {
  // Start
  hipblasCgemmBatched(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const cuComplex **/, a /*const cuComplex *const **/, lda /*int*/,
      b /*const cuComplex *const **/, ldb /*int*/, beta /*const cuComplex **/,
      c /*cuComplex *const **/, ldc /*int*/, group_count /*int*/);
  // End
}
