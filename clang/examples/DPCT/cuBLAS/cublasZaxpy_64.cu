
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *alpha,
          const hipDoubleComplex *x, int64_t incx, hipDoubleComplex *y,
          int64_t incy) {
  // Start
  hipblasZaxpy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 x /*const cuDoubleComplex **/, incx /*int64_t*/,
                 y /*cuDoubleComplex **/, incy /*int64_t*/);
  // End
}
