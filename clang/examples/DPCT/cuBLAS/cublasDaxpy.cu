
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *alpha, const double *x,
          int incx, double *y, int incy) {
  // Start
  hipblasDaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
              x /*const double **/, incx /*int*/, y /*const double **/,
              incy /*int*/);
  // End
}
