
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, float *x, int incx, float *y,
          int incy) {
  // Start
  hipblasSswap(handle /*cublasHandle_t*/, n /*int*/, x /*float **/, incx /*int*/,
              y /*float **/, incy /*int*/);
  // End
}
