
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const hipDoubleComplex *alpha,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, beta /*const cuDoubleComplex **/,
              c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
