
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const float *x, int incx, int *res) {
  // Start
  hipblasIsamin(h /*cublasHandle_t*/, n /*int*/, x /*const float **/,
               incx /*int*/, res /*int **/);
  // End
}
