
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const float *alpha, const hipComplex *x, int incx, hipComplex *a) {
  // Start
  hipblasChpr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const float **/, x /*const cuComplex **/,
             incx /*int*/, a /*cuComplex **/);
  // End
}
