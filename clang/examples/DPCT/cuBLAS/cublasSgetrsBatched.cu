
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const float *const *a, int lda, const int *ipiv, float *const *b,
          int ldb, int *info, int group_count) {
  // Start
  hipblasSgetrsBatched(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/,
                      n /*int*/, nrhs /*int*/, a /*const float *const **/,
                      lda /*int*/, ipiv /*const int **/, b /*float *const **/,
                      ldb /*int*/, info /*int **/, group_count /*int*/);
  // End
}
