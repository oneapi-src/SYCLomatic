
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const double *alpha,
          const double *a, int lda, double *b, int ldb) {
  // Start
  hipblasDtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const double **/, a /*const double **/, lda /*int*/,
              b /*double **/, ldb /*int*/);
  // End
}
