
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          const hipComplex *a, int lda, const hipComplex *x, int incx,
          hipComplex *c, int ldc) {
  // Start
  hipblasCdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              m /*int*/, n /*int*/, a /*const cuComplex **/, lda /*int*/,
              x /*const cuComplex **/, incx /*int*/, c /*cuComplex **/,
              ldc /*int*/);
  // End
}
