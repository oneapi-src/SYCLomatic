
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const double *x, int incx, double *res) {
  // Start
  hipblasDasum(h /*cublasHandle_t*/, n /*int*/, x /*const double **/,
              incx /*int*/, res /*double* */);
  // End
}
