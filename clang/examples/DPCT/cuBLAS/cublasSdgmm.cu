
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int m, int n,
          const float *a, int lda, const float *x, int incx, float *c,
          int ldc) {
  // Start
  hipblasSdgmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              m /*int*/, n /*int*/, a /*const float **/, lda /*int*/,
              x /*const float **/, incx /*int*/, c /*float **/, ldc /*int*/);
  // End
}
