
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          const float *beta, hipComplex *c, int ldc) {
  // Start
  hipblasCherkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const cuComplex **/, a /*const cuComplex **/,
               lda /*int*/, b /*const cuComplex **/, ldb /*int*/,
               beta /*const float **/, c /*cuComplex **/, ldc /*int*/);
  // End
}
