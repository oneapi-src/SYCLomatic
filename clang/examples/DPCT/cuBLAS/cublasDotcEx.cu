
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const void *x, hipDataType xtype,
          int incx, const void *y, hipDataType ytype, int incy, void *res,
          hipDataType restype, hipDataType computetype) {
  // Start
  hipblasDotcEx(handle /*cublasHandle_t*/, n /*int*/, x /*const void **/,
               xtype /*cudaDataType*/, incx /*int*/, y /*const void **/,
               ytype /*cudaDataType*/, incy /*int*/, res /*void **/,
               restype /*cudaDataType*/, computetype /*cudaDataType*/);
  // End
}
