
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(int n, int elementsize, const void *x, int incx, void *y, int incy) {
  // Start
  hipblasSetVector(n /*int*/, elementsize /*int*/, x /*const void **/,
                  incx /*int*/, y /*void **/, incy /*int*/);
  // End
}
