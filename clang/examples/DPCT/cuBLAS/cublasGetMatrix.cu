
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(int rows, int cols, int elementsize, const void *a, int lda, void *b,
          int ldb) {
  // Start
  hipblasGetMatrix(rows /*int*/, cols /*int*/, elementsize /*int*/,
                  a /*const void **/, lda /*int*/, b /*void **/, ldb /*int*/);
  // End
}
