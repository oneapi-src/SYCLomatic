
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipDoubleComplex *x, int incx,
          hipDoubleComplex *y, int incy, const double *c,
          const hipDoubleComplex *s) {
  // Start
  hipblasZrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
             incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
             c /*const double **/, s /*const cuDoubleComplex **/);
  // End
}
