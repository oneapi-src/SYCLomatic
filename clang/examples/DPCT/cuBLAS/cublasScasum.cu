
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *x, int incx,
          float *res) {
  // Start
  hipblasScasum(handle /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
               incx /*int*/, res /*float **/);
  // End
}
