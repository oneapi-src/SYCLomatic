
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const hipComplex *alpha,
          const hipComplex *a, int lda, hipComplex *b, int ldb) {
  // Start
  hipblasCtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
              b /*cuComplex **/, ldb /*int*/);
  // End
}
