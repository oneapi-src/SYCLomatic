
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const double *alpha, const double *x, int incx, double *a) {
  // Start
  hipblasDspr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const double **/, x /*const double **/,
             incx /*int*/, a /*double **/);
  // End
}
