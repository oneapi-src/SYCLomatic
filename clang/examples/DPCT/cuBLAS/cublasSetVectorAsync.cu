
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(int n, int elementsize, const void *from, int incx, void *to,
          int incy, hipStream_t stream) {
  // Start
  hipblasSetVectorAsync(n /*int*/, elementsize /*int*/, from /*const void **/,
                       incx /*int*/, to /*void **/, incy /*int*/,
                       stream /*cudaStream_t*/);
  // End
}
