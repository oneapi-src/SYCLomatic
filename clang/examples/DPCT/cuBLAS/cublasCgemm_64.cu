
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n, int64_t k,
          const hipComplex *alpha, const hipComplex *a, int64_t lda,
          const hipComplex *b, int64_t ldb, const hipComplex *beta, hipComplex *c,
          int64_t ldc) {
  // Start
  hipblasCgemm_64(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/, k /*int64_t*/,
      alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int64_t*/,
      b /*const cuComplex **/, ldb /*int64_t*/, beta /*const cuComplex **/,
      c /*cuComplex **/, ldc /*int64_t*/);
  // End
}
