
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, float *d1, float *d2, float *x1,
          const float *y1, float *param) {
  // Start
  hipblasSrotmg(handle /*cublasHandle_t*/, d1 /*float **/, d2 /*float **/,
               x1 /*float **/, y1 /*const float **/, param /*float**/);
  // End
}
