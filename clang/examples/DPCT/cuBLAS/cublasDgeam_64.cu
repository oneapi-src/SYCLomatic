
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n, const double *alpha,
          const double *a, int64_t lda, const double *beta, const double *b,
          int64_t ldb, double *c, int64_t ldc) {
  // Start
  hipblasDgeam_64(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                 transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const double **/, a /*const double **/,
                 lda /*int64_t*/, beta /*const double **/, b /*const double **/,
                 ldb /*int64_t*/, c /*double **/, ldc /*int64_t*/);
  // End
}
