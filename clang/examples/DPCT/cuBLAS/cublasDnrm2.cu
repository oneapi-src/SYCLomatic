
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *x, int incx,
          double *res) {
  // Start
  hipblasDnrm2(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
              incx /*int*/, res /*double **/);
  // End
}
