
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *x, int incx,
          double *res) {
  // Start
  hipblasDzasum(handle /*cublasHandle_t*/, n /*int*/,
               x /*const cuDoubleComplex **/, incx /*int*/, res /*double **/);
  // End
}
