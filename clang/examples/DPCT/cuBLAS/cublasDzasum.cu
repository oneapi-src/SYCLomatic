
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const hipDoubleComplex *x, int incx,
          double *res) {
  // Start
  hipblasDzasum(h /*cublasHandle_t*/, n /*int*/, x /*const cuDoubleComplex **/,
               incx /*int*/, res /*double* */);
  // End
}
