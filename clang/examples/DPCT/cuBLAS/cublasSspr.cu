
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const float *alpha, const float *x, int incx, float *a) {
  // Start
  hipblasSspr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const float **/, x /*const float **/,
             incx /*int*/, a /*float**/);
  // End
}
