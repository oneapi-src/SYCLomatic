
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const hipComplex *x, int incx, int *res) {
  // Start
  hipblasIcamin(h /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
               incx /*int*/, res /*int **/);
  // End
}
