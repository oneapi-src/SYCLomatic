
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const hipComplex *alpha,
          const hipComplex *a, int lda, long long int stridea,
          const hipComplex *b, int ldb, long long int strideb,
          const hipComplex *beta, hipComplex *c, int ldc, long long int stridec,
          int group_count) {
  // Start
  hipblasCgemmStridedBatched(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
      stridea /*long long int*/, b /*const cuComplex **/, ldb /*int*/,
      strideb /*long long int*/, beta /*const cuComplex **/, c /*cuComplex **/,
      ldc /*int*/, stridec /*long long int*/, group_count /*int*/);
  // End
}
