// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *x, int64_t incx,
          int64_t *res) {
  // Start
  hipblasIdamax_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  x /*const double **/, incx /*int64_t*/, res /*int64_t **/);
  // End
}
