
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const float *alpha, const hipComplex *x, int incx, hipComplex *a,
          int lda) {
  // Start
  hipblasCher(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const cuComplex **/, x /*const cuComplex **/,
             incx /*int*/, a /*const cuComplex **/, lda /*int*/);
  // End
}
