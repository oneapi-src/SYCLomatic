
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const double *alpha, const double *x, int incx, double *a, int lda) {
  // Start
  hipblasDsyr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const double **/, x /*const double **/,
             incx /*int*/, a /*double **/, lda /*int*/);
  // End
}
