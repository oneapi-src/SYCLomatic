
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int64_t n,
          const float *a, int64_t lda, float *x, int64_t incx) {
  // Start
  hipblasStrsv_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
                 n /*int64_t*/, a /*const float **/, lda /*int64_t*/,
                 x /*float **/, incx /*int64_t*/);
  // End
}
