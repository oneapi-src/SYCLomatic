
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipDoubleComplex *x, int incx,
          hipDoubleComplex *y, int incy, const double *c, const double *s) {
  // Start
  hipblasZdrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
              incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/,
              c /*const double **/, s /*const double **/);
  // End
}
