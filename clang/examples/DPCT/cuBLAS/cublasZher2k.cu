
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const hipDoubleComplex *alpha,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *b, int ldb,
          const double *beta, hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZher2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
               lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
               beta /*const double **/, c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
