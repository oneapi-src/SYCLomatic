
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int64_t m, int64_t n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          hipDoubleComplex *b, int64_t ldb) {
  // Start
  hipblasZtrsm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
                 unit_diag /*cublasDiagType_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 b /*cuDoubleComplex **/, ldb /*int64_t*/);
  // End
}
