
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *x, int incx,
          const float *y, int incy, float *res) {
  // Start
  hipblasSdot(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
             incx /*int*/, y /*const float **/, incy /*int*/, res /*float **/);
  // End
}
