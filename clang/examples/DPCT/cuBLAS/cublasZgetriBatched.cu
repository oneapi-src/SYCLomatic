
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *const *a,
          int lda, const int *ipiv, hipDoubleComplex *const *c, int ldc,
          int *info, int group_count) {
  // Start
  hipblasZgetriBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*const cuDoubleComplex *const **/, lda /*int*/,
                      ipiv /*const int **/, c /*cuDoubleComplex *const **/,
                      ldc /*int*/, info /*int **/, group_count /*int*/);
  // End
}
