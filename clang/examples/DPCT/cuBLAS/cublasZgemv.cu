
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *x, int incx, const hipDoubleComplex *beta,
          hipDoubleComplex *y, int incy) {
  // Start
  hipblasZgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, alpha /*const cuDoubleComplex **/,
              a /*const cuDoubleComplex **/, lda /*int*/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
              incy /*int*/);
  // End
}
