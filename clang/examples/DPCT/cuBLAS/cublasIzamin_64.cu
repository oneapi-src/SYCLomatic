// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *x,
          int64_t incx, int64_t *res) {
  // Start
  hipblasIzamin_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  x /*const cuDoubleComplex **/, incx /*int64_t*/,
                  res /*int64_t **/);
  // End
}
