
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const float *alpha, const float *a, int lda, const float *x, int incx,
          const float *beta, float *y, int incy) {
  // Start
  hipblasSsymv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const float **/, a /*const float **/,
              lda /*int*/, x /*const float **/, incx /*int*/,
              beta /*const float **/, y /*float **/, incy /*int*/);
  // End
}
