
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int64_t m, int64_t n, const double *alpha,
          const double *a, int64_t lda, double *b, int64_t ldb) {
  // Start
  hipblasDtrsm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
                 unit_diag /*cublasDiagType_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const double **/, a /*const double **/,
                 lda /*int64_t*/, b /*double **/, ldb /*int64_t*/);
  // End
}
