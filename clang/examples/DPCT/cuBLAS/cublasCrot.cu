
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipComplex *x, int incx, hipComplex *y,
          int incy, const float *c, const hipComplex *s) {
  // Start
  hipblasCrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuComplex **/,
             incx /*int*/, y /*cuComplex **/, incy /*int*/, c /*const float **/,
             s /*const cuComplex **/);
  // End
}
