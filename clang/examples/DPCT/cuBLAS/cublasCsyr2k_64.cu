
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const hipComplex *alpha,
          const hipComplex *a, int64_t lda, const hipComplex *b, int64_t ldb,
          const hipComplex *beta, hipComplex *c, int64_t ldc) {
  // Start
  hipblasCsyr2k_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                  trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                  alpha /*const cuComplex **/, a /*const cuComplex **/,
                  lda /*int64_t*/, b /*const cuComplex **/, ldb /*int64_t*/,
                  beta /*const cuComplex **/, c /*cuComplex **/,
                  ldc /*int64_t*/);
  // End
}
