
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, double *d1, double *d2, double *x1,
          const double *y1, double *param) {
  // Start
  hipblasDrotmg(handle /*cublasHandle_t*/, d1 /*double **/, d2 /*double **/,
               x1 /*double **/, y1 /*const double **/, param /*double **/);
  // End
}
