
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *x, int incx,
          hipDoubleComplex *a, int lda) {
  // Start
  hipblasZsyr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const cuDoubleComplex **/,
             x /*const cuDoubleComplex **/, incx /*int*/,
             a /*const cuDoubleComplex **/, lda /*int*/);
  // End
}
