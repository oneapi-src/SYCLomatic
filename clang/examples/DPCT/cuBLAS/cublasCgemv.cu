
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
          const hipComplex *alpha, const hipComplex *a, int lda,
          const hipComplex *x, int incx, const hipComplex *beta, hipComplex *y,
          int incy) {
  // Start
  hipblasCgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, alpha /*const cuComplex **/, a /*const cuComplex **/,
              lda /*int*/, x /*const cuComplex **/, incx /*int*/,
              beta /*const cuComplex **/, y /*cuComplex **/, incy /*int*/);
  // End
}
