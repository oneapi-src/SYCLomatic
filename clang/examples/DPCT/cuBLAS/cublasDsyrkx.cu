
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const double *alpha,
          const double *a, int lda, const double *b, int ldb,
          const double *beta, double *c, int ldc) {
  // Start
  hipblasDsyrkx(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const double **/, a /*const double **/, lda /*int*/,
               b /*const double **/, ldb /*int*/, beta /*const double **/,
               c /*double **/, ldc /*int*/);
  // End
}
