
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *alpha, hipComplex *x,
          int incx) {
  // Start
  hipblasCsscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const float **/,
               x /*cuComplex **/, incx /*int*/);
  // End
}
