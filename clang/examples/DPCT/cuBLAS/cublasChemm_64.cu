
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int64_t m, int64_t n,
          const hipComplex *alpha, const hipComplex *a, int64_t lda,
          const hipComplex *b, int64_t ldb, const hipComplex *beta, hipComplex *c,
          int64_t ldc) {
  // Start
  hipblasChemm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuComplex **/, a /*const cuComplex **/,
                 lda /*int64_t*/, b /*const cuComplex **/, ldb /*int64_t*/,
                 beta /*const cuComplex **/, c /*cuComplex **/,
                 ldc /*int64_t*/);
  // End
}
