
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipComplex *x, int incx, hipComplex *y,
          int incy, const float *c, const float *s) {
  // Start
  hipblasCsrot(handle /*cublasHandle_t*/, n /*int*/, x /*cuComplex **/,
              incx /*int*/, y /*cuComplex **/, incy /*int*/,
              c /*const float **/, s /*const float **/);
  // End
}
