
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl,
          int ku, const hipDoubleComplex *alpha, const hipDoubleComplex *a,
          int lda, const hipDoubleComplex *x, int incx,
          const hipDoubleComplex *beta, hipDoubleComplex *y, int incy) {
  // Start
  hipblasZgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, kl /*int*/, ku /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, x /*const cuDoubleComplex **/, incx /*int*/,
              beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
              incy /*int*/);
  // End
}
