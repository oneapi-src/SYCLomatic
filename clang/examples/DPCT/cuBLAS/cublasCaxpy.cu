
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *alpha,
          const hipComplex *x, int incx, hipComplex *y, int incy) {
  // Start
  hipblasCaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const cuComplex **/,
              x /*const cuComplex **/, incx /*int*/, y /*cuComplex **/,
              incy /*int*/);
  // End
}
