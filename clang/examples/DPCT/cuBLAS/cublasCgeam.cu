
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *beta,
          const hipComplex *b, int ldb, hipComplex *c, int ldc) {
  // Start
  hipblasCgeam(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
              transb /*cublasOperation_t*/, m /*int*/, n /*int*/,
              alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
              beta /*const cuComplex **/, b /*const cuComplex **/, ldb /*int*/,
              c /*cuComplex **/, ldc /*int*/);
  // End
}
