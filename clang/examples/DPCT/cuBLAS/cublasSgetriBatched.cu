
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *const *a, int lda,
          const int *ipiv, float *const *c, int ldc, int *info,
          int group_count) {
  // Start
  hipblasSgetriBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*const float *const **/, lda /*int*/,
                      ipiv /*const int **/, c /*float *const **/, ldc /*int*/,
                      info /*int **/, group_count /*int*/);
  // End
}
