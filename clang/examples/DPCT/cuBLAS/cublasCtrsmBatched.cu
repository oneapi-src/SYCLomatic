
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const hipComplex *alpha,
          const hipComplex *const *a, int lda, hipComplex *const *b, int ldb,
          int group_count) {
  // Start
  hipblasCtrsmBatched(
      handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
      upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
      unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
      alpha /*const cuComplex **/, a /*const cuComplex *const **/, lda /*int*/,
      b /*cuComplex *const **/, ldb /*int*/, group_count /*int*/);
  // End
}
