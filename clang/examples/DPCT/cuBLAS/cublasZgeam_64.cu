
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *beta, const hipDoubleComplex *b, int64_t ldb,
          hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZgeam_64(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                 transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 beta /*const cuDoubleComplex **/,
                 b /*const cuDoubleComplex **/, ldb /*int64_t*/,
                 c /*cuDoubleComplex **/, ldc /*int64_t*/);
  // End
}
