// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *x, int incx,
          int *res) {
  // Start
  hipblasIzamin(handle /*cublasHandle_t*/, n /*int*/,
               x /*const cuDoubleComplex **/, incx /*int*/, res /*int **/);
  // End
}
