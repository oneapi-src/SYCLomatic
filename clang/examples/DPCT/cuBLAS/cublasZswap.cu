
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipDoubleComplex *x, int incx,
          hipDoubleComplex *y, int incy) {
  // Start
  hipblasZswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuDoubleComplex **/,
              incx /*int*/, y /*cuDoubleComplex **/, incy /*int*/);
  // End
}
