
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(int n, int elementsize, const void *x, int incx, void *y, int incy) {
  // Start
  hipblasGetVector(n /*int*/, elementsize /*int*/, x /*const void **/,
                  incx /*int*/, y /*void **/, incy /*int*/);
  // End
}
