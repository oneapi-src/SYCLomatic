
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const double *alpha,
          const hipDoubleComplex *a, int64_t lda, const double *beta,
          hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZherk_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                 alpha /*const double **/, a /*const cuDoubleComplex **/,
                 lda /*int64_t*/, beta /*const double **/,
                 c /*cuDoubleComplex **/, ldc /*int64_t*/);
  // End
}
