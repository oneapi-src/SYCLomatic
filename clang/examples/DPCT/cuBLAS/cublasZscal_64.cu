
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *alpha,
          hipDoubleComplex *x, int64_t incx) {
  // Start
  hipblasZscal_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const cuDoubleComplex **/, x /*cuDoubleComplex **/,
                 incx /*int64_t*/);
  // End
}
