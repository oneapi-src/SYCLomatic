
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int64_t m, int64_t n, const float *alpha,
          const float *a, int64_t lda, float *b, int64_t ldb) {
  // Start
  hipblasStrsm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
                 unit_diag /*cublasDiagType_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const float **/, a /*const float **/, lda /*int64_t*/,
                 b /*float **/, ldb /*int64_t*/);
  // End
}
