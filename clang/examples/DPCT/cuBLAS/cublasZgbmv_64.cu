
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int64_t m, int64_t n,
          int64_t kl, int64_t ku, const hipDoubleComplex *alpha,
          const hipDoubleComplex *a, int64_t lda, const hipDoubleComplex *x,
          int64_t incx, const hipDoubleComplex *beta, hipDoubleComplex *y,
          int64_t incy) {
  // Start
  hipblasZgbmv_64(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/,
                 m /*int64_t*/, n /*int64_t*/, kl /*int64_t*/, ku /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 x /*const cuDoubleComplex **/, incx /*int64_t*/,
                 beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/,
                 incy /*int64_t*/);
  // End
}
