
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, hipComplex *x, int64_t incx,
          hipComplex *y, int64_t incy) {
  // Start
  hipblasCswap_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*cuComplex **/,
                 incx /*int64_t*/, y /*cuComplex **/, incy /*int64_t*/);
  // End
}
