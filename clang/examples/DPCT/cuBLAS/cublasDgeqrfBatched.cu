
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int m, int n, double *const *a, int lda,
          double *const *tau, int *info, int group_count) {
  // Start
  hipblasDgeqrfBatched(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
                      a /*double *const **/, lda /*int*/,
                      tau /*double *const **/, info /*int **/,
                      group_count /*int*/);
  // End
}
