
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *x, int64_t incx,
          double *y, int64_t incy) {
  // Start
  hipblasDcopy_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*const double **/,
                 incx /*int64_t*/, y /*double **/, incy /*int64_t*/);
  // End
}
