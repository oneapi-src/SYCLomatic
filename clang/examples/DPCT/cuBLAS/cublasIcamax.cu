// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *x, int incx,
          int *res) {
  // Start
  hipblasIcamax(handle /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
               incx /*int*/, res /*int **/);
  // End
}
