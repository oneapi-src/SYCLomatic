
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *x, int64_t incx,
          const hipComplex *y, int64_t incy, hipComplex *res) {
  // Start
  hipblasCdotu_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 x /*const cuComplex **/, incx /*int64_t*/,
                 y /*const cuComplex **/, incy /*int64_t*/,
                 res /*cuComplex **/);
  // End
}
