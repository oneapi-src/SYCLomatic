
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, float *const *a, int lda, int *ipiv,
          int *info, int group_count) {
  // Start
  hipblasSgetrfBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*float *const **/, lda /*int*/, ipiv /*int **/,
                      info /*int **/, group_count /*int*/);
  // End
}
