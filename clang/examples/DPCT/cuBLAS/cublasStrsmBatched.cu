
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const float *alpha,
          const float *const *a, int lda, float *const *b, int ldb,
          int group_count) {
  // Start
  hipblasStrsmBatched(
      handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
      upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
      unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
      alpha /*const float **/, a /*const float *const **/, lda /*int*/,
      b /*float *const **/, ldb /*int*/, group_count /*int*/);
  // End
}
