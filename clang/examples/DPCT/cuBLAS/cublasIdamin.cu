// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *x, int incx, int *res) {
  // Start
  hipblasIdamin(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
               incx /*int*/, res /*int **/);
  // End
}
