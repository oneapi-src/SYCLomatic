
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, hipDoubleComplex *x, int64_t incx,
          hipDoubleComplex *y, int64_t incy, const double *c,
          const hipDoubleComplex *s) {
  // Start
  hipblasZrot_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                x /*cuDoubleComplex **/, incx /*int64_t*/,
                y /*cuDoubleComplex **/, incy /*int64_t*/, c /*const double **/,
                s /*const cuDoubleComplex **/);
  // End
}
