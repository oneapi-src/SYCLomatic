
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const float *x, int incx, float *res) {
  // Start
  hipblasSasum(h /*cublasHandle_t*/, n /*int*/, x /*const float **/,
              incx /*int*/, res /*float* */);
  // End
}
