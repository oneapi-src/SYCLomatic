
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int m, int n, hipDoubleComplex *const *a,
          int lda, hipDoubleComplex *const *tau, int *info, int group_count) {
  // Start
  hipblasZgeqrfBatched(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
                      a /*cuDoubleComplex *const **/, lda /*int*/,
                      tau /*cuDoubleComplex *const **/, info /*int **/,
                      group_count /*int*/);
  // End
}
