
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *alpha, const float *x,
          int incx, float *y, int incy) {
  // Start
  hipblasSaxpy(handle /*cublasHandle_t*/, n /*int*/, alpha /*const float **/,
              x /*const float **/, incx /*int*/, y /*const float **/,
              incy /*int*/);
  // End
}
