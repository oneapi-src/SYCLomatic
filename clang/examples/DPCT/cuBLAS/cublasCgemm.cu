
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          const hipComplex *beta, hipComplex *c, int ldc) {
  // Start
  hipblasCgemm(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
              transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
              alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
              b /*const cuComplex **/, ldb /*int*/, beta /*const cuComplex **/,
              c /*cuComplex **/, ldc /*int*/);
  // End
}
