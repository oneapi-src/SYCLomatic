
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const double *alpha, const double *a, const double *x, int incx,
          const double *beta, double *y, int incy) {
  // Start
  hipblasDspmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const double **/, a /*const double **/,
              x /*const double **/, incx /*int*/, beta /*const double **/,
              y /*double **/, incy /*int*/);
  // End
}
