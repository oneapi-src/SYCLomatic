
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const float *alpha,
          const void *a, hipDataType atype, int lda, const void *b,
          hipDataType btype, int ldb, const float *beta, void *c,
          hipDataType ctype, int ldc) {
  // Start
  cublasSgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
                alpha /*const float **/, a /*const void **/,
                atype /*cudaDataType*/, lda /*int*/, b /*const void **/,
                btype /*cudaDataType*/, ldb /*int*/, beta /*const float **/,
                c /*void **/, ctype /*cudaDataType*/, ldc /*int*/);
  // End
}
