
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n, int64_t k,
          const float *alpha, const float *a, int64_t lda, const float *b,
          int64_t ldb, const float *beta, float *c, int64_t ldc) {
  // Start
  hipblasSgemm_64(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                 transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/,
                 k /*int64_t*/, alpha /*const float **/, a /*const float **/,
                 lda /*int64_t*/, b /*const float **/, ldb /*int64_t*/,
                 beta /*const float **/, c /*float **/, ldc /*int64_t*/);
  // End
}
