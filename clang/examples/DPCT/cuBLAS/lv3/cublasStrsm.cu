
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const float *alpha,
          const float *a, int lda, float *b, int ldb) {
  // Start
  hipblasStrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const float **/, a /*const float **/, lda /*int*/,
              b /*float **/, ldb /*int*/);
  // End
}
