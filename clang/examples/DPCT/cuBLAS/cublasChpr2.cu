
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipComplex *alpha, const hipComplex *x, int incx,
          const hipComplex *y, int incy, hipComplex *a) {
  // Start
  hipblasChpr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const cuComplex **/, x /*const cuComplex **/,
              incx /*int*/, y /*const cuComplex **/, incy /*int*/,
              a /*cuComplex **/);
  // End
}
