
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, float *x, int64_t incx, float *y,
          int64_t incy, const float *c, const float *s) {
  // Start
  hipblasSrot_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*float **/,
                incx /*int64_t*/, y /*float **/, incy /*int64_t*/,
                c /*const float **/, s /*const float **/);
  // End
}
