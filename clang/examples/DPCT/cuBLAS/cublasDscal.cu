
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *alpha, double *x,
          int incx) {
  // Start
  hipblasDscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
              x /*double **/, incx /*int*/);
  // End
}
