
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int64_t m, int64_t n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *b, int64_t ldb, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZsymm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 upper_lower /*cublasFillMode_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 b /*const cuDoubleComplex **/, ldb /*int64_t*/,
                 beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
                 ldc /*int64_t*/);
  // End
}
