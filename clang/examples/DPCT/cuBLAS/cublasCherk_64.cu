
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const float *alpha,
          const hipComplex *a, int64_t lda, const float *beta, hipComplex *c,
          int64_t ldc) {
  // Start
  hipblasCherk_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                 alpha /*const float **/, a /*const cuComplex **/,
                 lda /*int64_t*/, beta /*const float **/, c /*cuComplex **/,
                 ldc /*int64_t*/);
  // End
}
