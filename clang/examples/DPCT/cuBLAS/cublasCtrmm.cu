
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          hipComplex *c, int ldc) {
  // Start
  hipblasCtrmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
              b /*const cuComplex **/, ldb /*int*/, c /*cuComplex **/,
              ldc /*int*/);
  // End
}
