
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *x, int incx,
          hipDoubleComplex *y, int incy) {
  // Start
  hipblasZcopy(handle /*cublasHandle_t*/, n /*int*/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              y /*cuDoubleComplex **/, incy /*int*/);
  // End
}
