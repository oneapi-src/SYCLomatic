
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *alpha,
          hipDoubleComplex *x, int incx) {
  // Start
  hipblasZscal(handle /*cublasHandle_t*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, x /*cuDoubleComplex **/,
              incx /*int*/);
  // End
}
