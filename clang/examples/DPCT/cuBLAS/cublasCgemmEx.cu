
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const hipComplex *alpha,
          const void *a, hipDataType atype, int lda, const void *b,
          hipDataType btype, int ldb, const hipComplex *beta, void *c,
          hipDataType ctype, int ldc) {
  // Start
  cublasCgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
                alpha /*const cuComplex **/, a /*const void **/,
                atype /*cudaDataType_t*/, lda /*int*/, b /*const void **/,
                btype /*cudaDataType_t*/, ldb /*int*/,
                beta /*const cuComplex **/, c /*const void **/,
                ctype /*cudaDataType_t*/, ldc /*int*/);
  // End
}
