
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const hipComplex *alpha,
          const void *a, hipDataType atype, int lda, const void *b,
          hipDataType btype, int ldb, const hipComplex *beta, void *c,
          hipDataType ctype, int ldc) {
  // Start
  cublasCgemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
                alpha /*const cuComplex **/, a /*const void **/,
                atype /*cudaDataType*/, lda /*int*/, b /*const void **/,
                btype /*cudaDataType*/, ldb /*int*/, beta /*const cuComplex **/,
                c /*void **/, ctype /*cudaDataType*/, ldc /*int*/);
  // End
}
