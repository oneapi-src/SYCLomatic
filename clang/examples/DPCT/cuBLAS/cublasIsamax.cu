
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *x, int incx, int *res) {
  // Start
  hipblasIsamax(handle /*cublasHandle_t*/, n /*int*/, x /*const float **/,
               incx /*int*/, res /*int **/);
  // End
}
