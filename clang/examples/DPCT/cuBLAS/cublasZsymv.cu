
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *x, int incx, const hipDoubleComplex *beta,
          hipDoubleComplex *y, int incy) {
  // Start
  hipblasZsymv(
      handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/, n /*int*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
      lda /*int*/, x /*const cuDoubleComplex **/, incx /*int*/,
      beta /*const cuDoubleComplex **/, y /*cuDoubleComplex **/, incy /*int*/);
  // End
}
