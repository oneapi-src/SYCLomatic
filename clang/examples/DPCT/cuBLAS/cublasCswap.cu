
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, hipComplex *x, int incx, hipComplex *y,
          int incy) {
  // Start
  hipblasCswap(handle /*cublasHandle_t*/, n /*int*/, x /*cuComplex **/,
              incx /*int*/, y /*cuComplex **/, incy /*int*/);
  // End
}
