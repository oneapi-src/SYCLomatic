
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n,
          const hipComplex *alpha, const hipComplex *a, int64_t lda,
          const hipComplex *beta, const hipComplex *b, int64_t ldb, hipComplex *c,
          int64_t ldc) {
  // Start
  hipblasCgeam_64(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                 transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/,
                 alpha /*const cuComplex **/, a /*const cuComplex **/,
                 lda /*int64_t*/, beta /*const cuComplex **/,
                 b /*const cuComplex **/, ldb /*int64_t*/, c /*cuComplex **/,
                 ldc /*int64_t*/);
  // End
}
