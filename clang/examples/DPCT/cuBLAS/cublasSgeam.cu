
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, const float *alpha,
          const float *a, int lda, const float *beta, const float *b, int ldb,
          float *c, int ldc) {
  // Start
  hipblasSgeam(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
              transb /*cublasOperation_t*/, m /*int*/, n /*int*/,
              alpha /*const float **/, a /*const float **/, lda /*int*/,
              beta /*const float **/, b /*const float **/, ldb /*int*/,
              c /*float **/, ldc /*int*/);
  // End
}
