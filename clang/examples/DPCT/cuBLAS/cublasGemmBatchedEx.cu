
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const void *alpha,
          const void *const *a, hipDataType atype, int lda,
          const void *const *b, hipDataType btype, int ldb, const void *beta,
          void *const *c, hipDataType ctype, int ldc, int group_count,
          hipblasComputeType_t computetype_computeType_t,
          hipDataType computetype_dataType, hipblasGemmAlgo_t algo) {
  // Start
  hipblasGemmBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void *const **/, atype /*cudaDataType*/,
      lda /*int*/, b /*const void *const **/, btype /*cudaDataType*/,
      ldb /*int*/, beta /*const void **/, c /*void *const **/,
      ctype /*cudaDataType*/, ldc /*int*/, group_count /*int*/,
      computetype_computeType_t /*cublasComputeType_t*/,
      algo /*cublasGemmAlgo_t*/);
  hipblasGemmBatchedEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void *const **/, atype /*cudaDataType*/,
      lda /*int*/, b /*const void *const **/, btype /*cudaDataType*/,
      ldb /*int*/, beta /*const void **/, c /*void *const **/,
      ctype /*cudaDataType*/, ldc /*int*/, group_count /*int*/,
      computetype_dataType /*cudaDataType*/, algo /*cublasGemmAlgo_t*/);
  // End
}
