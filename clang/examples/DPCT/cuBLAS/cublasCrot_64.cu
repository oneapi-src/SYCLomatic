
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, hipComplex *x, int64_t incx,
          hipComplex *y, int64_t incy, const float *c, const hipComplex *s) {
  // Start
  hipblasCrot_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*cuComplex **/,
                incx /*int64_t*/, y /*cuComplex **/, incy /*int64_t*/,
                c /*const float **/, s /*const cuComplex **/);
  // End
}
