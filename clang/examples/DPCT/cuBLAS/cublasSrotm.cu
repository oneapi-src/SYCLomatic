
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, float *x, int incx, float *y, int incy,
          const float *param) {
  // Start
  hipblasSrotm(handle /*cublasHandle_t*/, n /*int*/, x /*float **/, incx /*int*/,
              y /*float **/, incy /*int*/, param /*const float**/);
  // End
}
