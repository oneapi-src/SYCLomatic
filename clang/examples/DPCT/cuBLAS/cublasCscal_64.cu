
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *alpha,
          hipComplex *x, int64_t incx) {
  // Start
  hipblasCscal_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const cuComplex **/, x /*cuComplex **/,
                 incx /*int64_t*/);
  // End
}
