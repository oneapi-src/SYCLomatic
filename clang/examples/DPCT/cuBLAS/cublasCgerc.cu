
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int m, int n, const hipComplex *alpha,
          const hipComplex *x, int incx, const hipComplex *y, int incy,
          hipComplex *a, int lda) {
  // Start
  hipblasCgerc(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
              alpha /*const cuComplex **/, x /*const cuComplex **/,
              incx /*int*/, y /*const cuComplex **/, incy /*int*/,
              a /*cuComplex **/, lda /*int*/);
  // End
}
