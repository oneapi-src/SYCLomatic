
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int64_t m,
          int64_t n, const float *a, int64_t lda, const float *x, int64_t incx,
          float *c, int64_t ldc) {
  // Start
  hipblasSdgmm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 m /*int64_t*/, n /*int64_t*/, a /*const float **/,
                 lda /*int64_t*/, x /*const float **/, incx /*int64_t*/,
                 c /*float **/, ldc /*int64_t*/);
  // End
}
