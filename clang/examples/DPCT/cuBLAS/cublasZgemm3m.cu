
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *b, int ldb, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int ldc) {
  // Start
  cublasZgemm3m(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
      lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
      beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
