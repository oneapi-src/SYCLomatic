
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const double *alpha, const hipDoubleComplex *x, int incx,
          hipDoubleComplex *a, int lda) {
  // Start
  hipblasZher(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const double **/, x /*const cuDoubleComplex **/,
             incx /*int*/, a /*cuDoubleComplex **/, lda /*int*/);
  // End
}
