
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int64_t m,
          int64_t n, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *x, int64_t incx, hipDoubleComplex *c,
          int64_t ldc) {
  // Start
  hipblasZdgmm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 m /*int64_t*/, n /*int64_t*/, a /*const cuDoubleComplex **/,
                 lda /*int64_t*/, x /*const cuDoubleComplex **/,
                 incx /*int64_t*/, c /*cuDoubleComplex **/, ldc /*int64_t*/);
  // End
}
