
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *x, int incx,
          const hipDoubleComplex *y, int incy, hipDoubleComplex *res) {
  // Start
  hipblasZdotu(handle /*cublasHandle_t*/, n /*int*/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              y /*const cuDoubleComplex **/, incy /*int*/,
              res /*cuDoubleComplex **/);
  // End
}
