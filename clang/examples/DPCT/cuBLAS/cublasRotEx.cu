
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, void *x, hipDataType xtype, int incx,
          void *y, hipDataType ytype, int incy, const void *c, const void *s,
          hipDataType cstype, hipDataType computetype) {
  // Start
  hipblasRotEx(handle /*cublasHandle_t*/, n /*int*/, x /*void **/,
              xtype /*cudaDataType*/, incx /*int*/, y /*void **/,
              ytype /*cudaDataType*/, incy /*int*/, c /*const void **/,
              s /*const void **/, cstype /*cudaDataType*/,
              computetype /*cudaDataType*/);
  // End
}
