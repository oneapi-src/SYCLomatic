
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *x,
          int64_t incx, double *res) {
  // Start
  hipblasDznrm2_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  x /*const cuDoubleComplex **/, incx /*int64_t*/,
                  res /*double **/);
  // End
}
