
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *alpha, hipDoubleComplex *x,
          int incx) {
  // Start
  hipblasZdscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const double **/,
               x /*cuDoubleComplex **/, incx /*int*/);
  // End
}
