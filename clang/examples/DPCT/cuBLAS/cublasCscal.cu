
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *alpha, hipComplex *x,
          int incx) {
  // Start
  hipblasCscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const cuComplex **/,
              x /*cuComplex **/, incx /*int*/);
  // End
}
