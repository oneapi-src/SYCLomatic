
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k, const void *alpha,
          const void *a, hipDataType atype, int lda, const void *b,
          hipDataType btype, int ldb, const void *beta, void *c,
          hipDataType ctype, int ldc,
          hipblasComputeType_t computetype_computeType_t,
          hipDataType computetype_dataType, hipblasGemmAlgo_t algo) {
  // Start
  hipblasGemmEx(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
               transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
               alpha /*const void **/, a /*const void **/,
               atype /*cudaDataType*/, lda /*int*/, b /*const void **/,
               btype /*cudaDataType*/, ldb /*int*/, beta /*const void **/,
               c /*void **/, ctype /*cudaDataType*/, ldc /*int*/,
               computetype_computeType_t /*cublasComputeType_t*/,
               algo /*cublasGemmAlgo_t*/);
  hipblasGemmEx(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const void **/, a /*const void **/, atype /*cudaDataType*/,
      lda /*int*/, b /*const void **/, btype /*cudaDataType*/, ldb /*int*/,
      beta /*const void **/, c /*void **/, ctype /*cudaDataType*/, ldc /*int*/,
      computetype_dataType /*cudaDataType*/, algo /*cublasGemmAlgo_t*/);
  // End
}
