
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *x,
          int64_t incx, hipDoubleComplex *y, int64_t incy) {
  // Start
  hipblasZcopy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 x /*const cuDoubleComplex **/, incx /*int64_t*/,
                 y /*cuDoubleComplex **/, incy /*int64_t*/);
  // End
}
