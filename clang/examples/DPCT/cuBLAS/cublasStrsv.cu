
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n,
          const float *a, int lda, float *x, int incx) {
  // Start
  hipblasStrsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, a /*const float **/, lda /*int*/, x /*float **/,
              incx /*int*/);
  // End
}
