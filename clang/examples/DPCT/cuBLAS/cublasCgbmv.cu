
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl,
          int ku, const hipComplex *alpha, const hipComplex *a, int lda,
          const hipComplex *x, int incx, const hipComplex *beta, hipComplex *y,
          int incy) {
  // Start
  hipblasCgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, kl /*int*/, ku /*int*/, alpha /*const cuComplex **/,
              a /*const cuComplex **/, lda /*int*/, x /*const cuComplex **/,
              incx /*int*/, beta /*const cuComplex **/, y /*cuComplex **/,
              incy /*int*/);
  // End
}
