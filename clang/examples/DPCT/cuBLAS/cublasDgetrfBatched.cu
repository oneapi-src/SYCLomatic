
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, double *const *a, int lda, int *ipiv,
          int *info, int group_count) {
  // Start
  hipblasDgetrfBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*double *const **/, lda /*int*/, ipiv /*int **/,
                      info /*int **/, group_count /*int*/);
  // End
}
