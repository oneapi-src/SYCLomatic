
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *x, int incx, double *y,
          int incy) {
  // Start
  hipblasDcopy(handle /*cublasHandle_t*/, n /*int*/, x /*const double **/,
              incx /*int*/, y /*double **/, incy /*int*/);
  // End
}
