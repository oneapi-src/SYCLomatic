
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *alpha,
          const hipComplex *x, int64_t incx, hipComplex *y, int64_t incy) {
  // Start
  hipblasCaxpy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const cuComplex **/, x /*const cuComplex **/,
                 incx /*int64_t*/, y /*cuComplex **/, incy /*int64_t*/);
  // End
}
