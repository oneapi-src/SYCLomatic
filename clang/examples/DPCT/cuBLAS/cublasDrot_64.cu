
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, double *x, int64_t incx, double *y,
          int64_t incy, const double *c, const double *s) {
  // Start
  hipblasDrot_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*double **/,
                incx /*int64_t*/, y /*double **/, incy /*int64_t*/,
                c /*const double **/, s /*const double **/);
  // End
}
