
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, const hipDoubleComplex *alpha,
          const hipDoubleComplex *a, int lda, const hipDoubleComplex *beta,
          const hipDoubleComplex *b, int ldb, hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZgeam(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
              transb /*cublasOperation_t*/, m /*int*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, beta /*const cuDoubleComplex **/,
              b /*const cuDoubleComplex **/, ldb /*int*/,
              c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
