
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const void *x, hipDataType xtype,
          int incx, void *res, hipDataType restype, hipDataType computetype) {
  // Start
  hipblasNrm2Ex(handle /*cublasHandle_t*/, n /*int*/, x /*const void **/,
               xtype /*cudaDataType*/, incx /*int*/, res /*void **/,
               restype /*cudaDataType*/, computetype /*cudaDataType*/);
  // End
}
