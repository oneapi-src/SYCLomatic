
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int64_t n,
          int64_t k, const hipComplex *a, int64_t lda, hipComplex *x,
          int64_t incx) {
  // Start
  hipblasCtbmv_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
                 n /*int64_t*/, k /*int64_t*/, a /*const cuComplex **/,
                 lda /*int64_t*/, x /*cuComplex **/, incx /*int64_t*/);
  // End
}
