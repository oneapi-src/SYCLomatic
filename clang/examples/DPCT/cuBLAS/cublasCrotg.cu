
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipComplex *a, hipComplex *b, float *c,
          hipComplex *s) {
  // Start
  hipblasCrotg(handle /*cublasHandle_t*/, a /*cuComplex **/, b /*cuComplex **/,
              c /*float **/, s /*cuComplex **/);
  // End
}
