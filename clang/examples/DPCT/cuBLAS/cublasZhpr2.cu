
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *x, int incx,
          const hipDoubleComplex *y, int incy, hipDoubleComplex *a) {
  // Start
  hipblasZhpr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const float **/, x /*const cuDoubleComplex **/,
              incx /*int*/, y /*const cuDoubleComplex **/, incy /*int*/,
              a /*cuDoubleComplex **/);
  // End
}
