
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const double *const *a, int lda,
          int *ipiv, double *const *c, int ldc, int *info, int group_count) {
  // Start
  hipblasDgetriBatched(handle /*cublasHandle_t*/, n /*int*/,
                      a /*const double *const **/, lda /*int*/, ipiv /*int **/,
                      c /*double *const **/, ldc /*int*/, info /*int **/,
                      group_count /*int*/);
  // End
}
