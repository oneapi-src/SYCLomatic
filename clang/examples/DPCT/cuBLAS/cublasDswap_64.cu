
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, double *x, int64_t incx, double *y,
          int64_t incy) {
  // Start
  hipblasDswap_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*double **/,
                 incx /*int64_t*/, y /*double **/, incy /*int64_t*/);
  // End
}
