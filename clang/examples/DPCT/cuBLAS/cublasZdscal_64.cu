
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const double *alpha,
          hipDoubleComplex *x, int64_t incx) {
  // Start
  hipblasZdscal_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  alpha /*const double **/, x /*cuDoubleComplex **/,
                  incx /*int64_t*/);
  // End
}
