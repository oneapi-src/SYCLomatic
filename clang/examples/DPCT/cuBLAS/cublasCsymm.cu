
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int m, int n, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          const hipComplex *beta, hipComplex *c, int ldc) {
  // Start
  hipblasCsymm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, m /*int*/, n /*int*/,
              alpha /*const cuComplex **/, a /*const cuComplex **/, lda /*int*/,
              b /*const cuComplex **/, ldb /*int*/, beta /*const cuComplex **/,
              c /*cuComplex **/, ldc /*int*/);
  // End
}
