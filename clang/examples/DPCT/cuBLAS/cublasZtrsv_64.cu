
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int64_t n,
          const hipDoubleComplex *a, int64_t lda, hipDoubleComplex *x,
          int64_t incx) {
  // Start
  hipblasZtrsv_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
                 n /*int64_t*/, a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 x /*cuDoubleComplex **/, incx /*int64_t*/);
  // End
}
