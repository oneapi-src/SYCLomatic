
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *x, int incx,
          const hipDoubleComplex *y, int incy, hipDoubleComplex *a, int lda) {
  // Start
  hipblasZsyr2(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              n /*int*/, alpha /*const cuDoubleComplex **/,
              x /*const cuDoubleComplex **/, incx /*int*/,
              y /*const cuDoubleComplex **/, incy /*int*/,
              a /*cuDoubleComplex **/, lda /*int*/);
  // End
}
