
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const void *alpha,
          hipDataType alphatype, const void *x, hipDataType xtype, int incx,
          void *y, hipDataType ytype, int incy, hipDataType computetype) {
  // Start
  hipblasAxpyEx(handle /*cublasHandle_t*/, n /*int*/, alpha /*const void **/,
               alphatype /*cudaDataType*/, x /*const void **/,
               xtype /*cudaDataType*/, incx /*int*/, y /*void **/,
               ytype /*cudaDataType*/, incy /*int*/,
               computetype /*cudaDataType*/);
  // End
}
