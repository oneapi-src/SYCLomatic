
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n,
          const double *a, int lda, double *x, int incx) {
  // Start
  hipblasDtrmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, a /*const double **/, lda /*int*/, x /*double **/,
              incx /*int*/);
  // End
}
