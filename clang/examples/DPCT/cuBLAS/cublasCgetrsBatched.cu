
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const hipComplex *const *a, int lda, const int *ipiv,
          hipComplex *const *b, int ldb, int *info, int group_count) {
  // Start
  hipblasCgetrsBatched(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/,
                      n /*int*/, nrhs /*int*/, a /*const cuComplex *const **/,
                      lda /*int*/, ipiv /*const int **/,
                      b /*cuComplex *const **/, ldb /*int*/, info /*int **/,
                      group_count /*int*/);
  // End
}
