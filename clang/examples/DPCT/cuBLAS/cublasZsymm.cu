
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, int m, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *b, int ldb, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZsymm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, m /*int*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
              beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
              ldc /*int*/);
  // End
}
