
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const double *alpha,
          const double *a, int lda, const double *beta, double *c, int ldc) {
  // Start
  hipblasDsyrk(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
              alpha /*const double **/, a /*const double **/, lda /*int*/,
              beta /*const double **/, c /*double **/, ldc /*int*/);
  // End
}
