
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          const hipDoubleComplex *b, int ldb, hipDoubleComplex *c, int ldc) {
  // Start
  hipblasZtrmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, b /*const cuDoubleComplex **/, ldb /*int*/,
              c /*cuDoubleComplex **/, ldc /*int*/);
  // End
}
