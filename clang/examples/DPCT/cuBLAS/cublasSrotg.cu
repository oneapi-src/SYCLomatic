
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, float *a, float *b, float *c, float *s) {
  // Start
  hipblasSrotg(handle /*cublasHandle_t*/, a /*float **/, b /*float **/,
              c /*float **/, s /*float **/);
  // End
}
