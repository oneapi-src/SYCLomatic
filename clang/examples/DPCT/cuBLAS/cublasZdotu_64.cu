
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipDoubleComplex *x,
          int64_t incx, const hipDoubleComplex *y, int64_t incy,
          hipDoubleComplex *res) {
  // Start
  hipblasZdotu_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 x /*const cuDoubleComplex **/, incx /*int64_t*/,
                 y /*const cuDoubleComplex **/, incy /*int64_t*/,
                 res /*cuDoubleComplex **/);
  // End
}
