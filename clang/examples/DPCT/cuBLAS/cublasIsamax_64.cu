// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const float *x, int64_t incx,
          int64_t *res) {
  // Start
  hipblasIsamax_64(handle /*cublasHandle_t*/, n /*int64_t*/, x /*const float **/,
                  incx /*int64_t*/, res /*int64_t **/);
  // End
}
