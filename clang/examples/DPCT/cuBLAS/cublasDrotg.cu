
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, double *a, double *b, double *c, double *s) {
  // Start
  hipblasDrotg(handle /*cublasHandle_t*/, a /*double **/, b /*double **/,
              c /*double **/, s /*double **/);
  // End
}
