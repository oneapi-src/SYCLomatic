
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipDoubleComplex *a, hipDoubleComplex *b,
          double *c, hipDoubleComplex *s) {
  // Start
  hipblasZrotg(handle /*cublasHandle_t*/, a /*cuDoubleComplex **/,
              b /*cuDoubleComplex **/, c /*double **/, s /*cuDoubleComplex **/);
  // End
}
