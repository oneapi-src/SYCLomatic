
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n, const double *alpha,
          const double *a, int lda, const double *b, int ldb, double *c,
          int ldc) {
  // Start
  hipblasDtrmm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const double **/, a /*const double **/, lda /*int*/,
              b /*const double **/, ldb /*int*/, c /*double **/, ldc /*int*/);
  // End
}
