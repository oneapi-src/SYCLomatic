
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *x, int64_t incx,
          float *res) {
  // Start
  hipblasScasum_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  x /*const cuComplex **/, incx /*int64_t*/, res /*float **/);
  // End
}
