
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipComplex *alpha, const hipComplex *x, int incx, hipComplex *a,
          int lda) {
  // Start
  hipblasCsyr(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
             n /*int*/, alpha /*const cuComplex **/, x /*const cuComplex **/,
             incx /*int*/, a /*cuComplex **/, lda /*int*/);
  // End
}
