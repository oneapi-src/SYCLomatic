
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int64_t n,
          const hipDoubleComplex *a, hipDoubleComplex *x, int64_t incx) {
  // Start
  hipblasZtpsv_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
                 n /*int64_t*/, a /*const cuDoubleComplex **/,
                 x /*cuDoubleComplex **/, incx /*int64_t*/);
  // End
}
