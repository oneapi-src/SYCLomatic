
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n, int kl,
          int ku, const float *alpha, const float *a, int lda, const float *x,
          int incx, const float *beta, float *y, int incy) {
  // Start
  hipblasSgbmv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, kl /*int*/, ku /*int*/, alpha /*const float **/,
              a /*const float **/, lda /*int*/, x /*const float **/,
              incx /*int*/, beta /*const float **/, y /*float **/,
              incy /*int*/);
  // End
}
