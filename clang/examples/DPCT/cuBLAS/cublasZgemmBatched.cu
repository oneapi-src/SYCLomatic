
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int m, int n, int k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *const *a,
          int lda, const hipDoubleComplex *const *b, int ldb,
          const hipDoubleComplex *beta, hipDoubleComplex *const *c, int ldc,
          int group_count) {
  // Start
  hipblasZgemmBatched(
      handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
      transb /*cublasOperation_t*/, m /*int*/, n /*int*/, k /*int*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex *const **/,
      lda /*int*/, b /*const cuDoubleComplex *const **/, ldb /*int*/,
      beta /*const cuDoubleComplex **/, c /*cuDoubleComplex *const **/,
      ldc /*int*/, group_count /*int*/);
  // End
}
