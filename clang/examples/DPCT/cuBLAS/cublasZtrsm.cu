
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int lda,
          hipDoubleComplex *b, int ldb) {
  // Start
  hipblasZtrsm(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
              upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
              unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex **/,
              lda /*int*/, b /*cuDoubleComplex **/, ldb /*int*/);
  // End
}
