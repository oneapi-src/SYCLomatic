
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right,
          hipblasFillMode_t upper_lower, hipblasOperation_t transa,
          hipblasDiagType_t unit_diag, int m, int n,
          const hipDoubleComplex *alpha, const hipDoubleComplex *const *a,
          int lda, hipDoubleComplex *const *b, int ldb, int group_count) {
  // Start
  hipblasZtrsmBatched(
      handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
      upper_lower /*cublasFillMode_t*/, transa /*cublasOperation_t*/,
      unit_diag /*cublasDiagType_t*/, m /*int*/, n /*int*/,
      alpha /*const cuDoubleComplex **/, a /*const cuDoubleComplex *const **/,
      lda /*int*/, b /*cuDoubleComplex *const **/, ldb /*int*/,
      group_count /*int*/);
  // End
}
