
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const double *alpha,
          const double *a, int64_t lda, const double *b, int64_t ldb,
          const double *beta, double *c, int64_t ldc) {
  // Start
  hipblasDsyr2k_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                  trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                  alpha /*const double **/, a /*const double **/,
                  lda /*int64_t*/, b /*const double **/, ldb /*int64_t*/,
                  beta /*const double **/, c /*double **/, ldc /*int64_t*/);
  // End
}
