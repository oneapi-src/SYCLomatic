
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *x, int incx,
          hipComplex *y, int incy) {
  // Start
  hipblasCcopy(handle /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
              incx /*int*/, y /*cuComplex **/, incy /*int*/);
  // End
}
