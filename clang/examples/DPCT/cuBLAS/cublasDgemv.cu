
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
          const double *alpha, const double *a, int lda, const double *x,
          int incx, const double *beta, double *y, int incy) {
  // Start
  hipblasDgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, alpha /*const double **/, a /*const double **/,
              lda /*int*/, x /*const double **/, incx /*int*/,
              beta /*const double **/, y /*double **/, incy /*int*/);
  // End
}
