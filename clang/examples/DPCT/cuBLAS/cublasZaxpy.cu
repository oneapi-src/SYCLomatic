
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *alpha,
          const hipDoubleComplex *x, int incx, hipDoubleComplex *y, int incy) {
  // Start
  hipblasZaxpy(handle /*cublasHandle_t*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, x /*const cuDoubleComplex **/,
              incx /*int*/, y /*const cuDoubleComplex **/, incy /*int*/);
  // End
}
