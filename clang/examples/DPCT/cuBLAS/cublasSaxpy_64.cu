
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const float *alpha, const float *x,
          int64_t incx, float *y, int64_t incy) {
  // Start
  hipblasSaxpy_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                 alpha /*const float **/, x /*const float **/, incx /*int64_t*/,
                 y /*float **/, incy /*int64_t*/);
  // End
}
