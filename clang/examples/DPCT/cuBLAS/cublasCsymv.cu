#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower, int n,
          const hipComplex *alpha, const hipComplex *a, int lda,
          const hipComplex *x, int incx, const hipComplex *beta, hipComplex *y,
          int incy) {
  // Start
  hipblasSsymv(handle /*hipblasHandle_t*/, upper_lower /*hipblasFillMode_t*/,
              n /*int*/, alpha /*const hipComplex **/, a /*const hipComplex **/,
              lda /*int*/, x /*const hipComplex **/, incx /*int*/,
              beta /*const hipComplex **/, y /*hipComplex **/, incy /*int*/);
  // End
}
