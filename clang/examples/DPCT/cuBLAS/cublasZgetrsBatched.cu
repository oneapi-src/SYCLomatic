
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int n, int nrhs,
          const hipDoubleComplex *const *a, int lda, const int *ipiv,
          hipDoubleComplex *const *b, int ldb, int *info, int group_count) {
  // Start
  hipblasZgetrsBatched(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/,
                      n /*int*/, nrhs /*int*/,
                      a /*const cuDoubleComplex *const **/, lda /*int*/,
                      ipiv /*const int **/, b /*cuDoubleComplex *const **/,
                      ldb /*int*/, info /*int **/, group_count /*int*/);
  // End
}
