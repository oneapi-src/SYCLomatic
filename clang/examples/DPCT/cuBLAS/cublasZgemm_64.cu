
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t transa,
          hipblasOperation_t transb, int64_t m, int64_t n, int64_t k,
          const hipDoubleComplex *alpha, const hipDoubleComplex *a, int64_t lda,
          const hipDoubleComplex *b, int64_t ldb, const hipDoubleComplex *beta,
          hipDoubleComplex *c, int64_t ldc) {
  // Start
  hipblasZgemm_64(handle /*cublasHandle_t*/, transa /*cublasOperation_t*/,
                 transb /*cublasOperation_t*/, m /*int64_t*/, n /*int64_t*/,
                 k /*int64_t*/, alpha /*const cuDoubleComplex **/,
                 a /*const cuDoubleComplex **/, lda /*int64_t*/,
                 b /*const cuDoubleComplex **/, ldb /*int64_t*/,
                 beta /*const cuDoubleComplex **/, c /*cuDoubleComplex **/,
                 ldc /*int64_t*/);
  // End
}
