
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int64_t m, int64_t n,
          const hipComplex *alpha, const hipComplex *a, int64_t lda,
          const hipComplex *x, int64_t incx, const hipComplex *beta, hipComplex *y,
          int64_t incy) {
  // Start
  hipblasCgemv_64(
      handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int64_t*/,
      n /*int64_t*/, alpha /*const cuComplex **/, a /*const cuComplex **/,
      lda /*int64_t*/, x /*const cuComplex **/, incx /*int64_t*/,
      beta /*const cuComplex **/, y /*cuComplex **/, incy /*int64_t*/);
  // End
}
