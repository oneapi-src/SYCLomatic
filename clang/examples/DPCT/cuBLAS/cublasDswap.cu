
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, double *x, int incx, double *y,
          int incy) {
  // Start
  hipblasDswap(handle /*cublasHandle_t*/, n /*int*/, x /*double **/,
              incx /*int*/, y /*double **/, incy /*int*/);
  // End
}
