
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int64_t n, int64_t k, const hipComplex *alpha,
          const hipComplex *a, int64_t lda, const hipComplex *beta, hipComplex *c,
          int64_t ldc) {
  // Start
  hipblasCsyrk_64(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
                 trans /*cublasOperation_t*/, n /*int64_t*/, k /*int64_t*/,
                 alpha /*const cuComplex **/, a /*const cuComplex **/,
                 lda /*int64_t*/, beta /*const cuComplex **/, c /*cuComplex **/,
                 ldc /*int64_t*/);
  // End
}
