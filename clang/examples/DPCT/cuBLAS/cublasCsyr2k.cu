
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, int n, int k, const hipComplex *alpha,
          const hipComplex *a, int lda, const hipComplex *b, int ldb,
          const hipComplex *beta, hipComplex *c, int ldc) {
  // Start
  hipblasCsyr2k(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
               trans /*cublasOperation_t*/, n /*int*/, k /*int*/,
               alpha /*const cuComplex **/, a /*const cuComplex **/,
               lda /*int*/, b /*const cuComplex **/, ldb /*int*/,
               beta /*const cuComplex **/, c /*cuComplex **/, ldc /*int*/);
  // End
}
