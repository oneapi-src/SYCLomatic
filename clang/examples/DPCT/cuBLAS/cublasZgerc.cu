
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int m, int n, const hipDoubleComplex *alpha,
          const hipDoubleComplex *x, int incx, const hipDoubleComplex *y,
          int incy, hipDoubleComplex *a, int lda) {
  // Start
  hipblasZgerc(handle /*cublasHandle_t*/, m /*int*/, n /*int*/,
              alpha /*const cuDoubleComplex **/, x /*const cuDoubleComplex **/,
              incx /*int*/, y /*const cuDoubleComplex **/, incy /*int*/,
              a /*cuDoubleComplex **/, lda /*int*/);
  // End
}
