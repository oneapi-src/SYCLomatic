
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipDoubleComplex *x, int incx,
          int *res) {
  // Start
  hipblasIzamax(handle /*cublasHandle_t*/, n /*int*/,
               x /*const cuDoubleComplex **/, incx /*int*/, res /*int **/);
  // End
}
