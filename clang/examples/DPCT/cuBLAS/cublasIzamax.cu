
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t h, int n, const hipDoubleComplex *x, int incx,
          int *res) {
  // Start
  hipblasIzamax(h /*cublasHandle_t*/, n /*int*/, x /*const cuDoubleComplex **/,
               incx /*int*/, res /*int **/);
  // End
}
