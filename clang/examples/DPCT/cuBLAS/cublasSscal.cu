
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const float *alpha, float *x,
          int incx) {
  // Start
  hipblasSscal(handle /*cublasHandle_t*/, n /*int*/, alpha /*const float **/,
              x /*float **/, incx /*int*/);
  // End
}
