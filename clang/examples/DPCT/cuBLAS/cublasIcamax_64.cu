// Option: --no-dry-pattern

#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int64_t n, const hipComplex *x, int64_t incx,
          int64_t *res) {
  // Start
  hipblasIcamax_64(handle /*cublasHandle_t*/, n /*int64_t*/,
                  x /*const cuComplex **/, incx /*int64_t*/, res /*int64_t **/);
  // End
}
