
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n, int k,
          const hipComplex *a, int lda, hipComplex *x, int incx) {
  // Start
  hipblasCtbsv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, k /*int*/, a /*const cuComplex **/, lda /*int*/,
              x /*cuComplex **/, incx /*int*/);
  // End
}
