
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasOperation_t trans, int m, int n,
          const float *alpha, const float *a, int lda, const float *x, int incx,
          const float *beta, float *y, int incy) {
  // Start
  hipblasSgemv(handle /*cublasHandle_t*/, trans /*cublasOperation_t*/, m /*int*/,
              n /*int*/, alpha /*const float **/, a /*const float **/,
              lda /*int*/, x /*const float **/, incx /*int*/,
              beta /*const float **/, y /*float **/, incy /*int*/);
  // End
}
