
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasFillMode_t upper_lower,
          hipblasOperation_t trans, hipblasDiagType_t unit_nonunit, int n, int k,
          const hipDoubleComplex *a, int lda, hipDoubleComplex *x, int incx) {
  // Start
  hipblasZtbmv(handle /*cublasHandle_t*/, upper_lower /*cublasFillMode_t*/,
              trans /*cublasOperation_t*/, unit_nonunit /*cublasDiagType_t*/,
              n /*int*/, k /*int*/, a /*const cuDoubleComplex **/, lda /*int*/,
              x /*cuDoubleComplex **/, incx /*int*/);
  // End
}
