
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, int n, const hipComplex *x, int incx,
          const hipComplex *y, int incy, hipComplex *res) {
  // Start
  hipblasCdotc(handle /*cublasHandle_t*/, n /*int*/, x /*const cuComplex **/,
              incx /*int*/, y /*const cuComplex **/, incy /*int*/,
              res /*cuComplex **/);
  // End
}
