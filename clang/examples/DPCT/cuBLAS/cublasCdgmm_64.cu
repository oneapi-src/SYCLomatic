
#include <hip/hip_runtime.h>
#include "hipblas.h"

void test(hipblasHandle_t handle, hipblasSideMode_t left_right, int64_t m,
          int64_t n, const hipComplex *a, int64_t lda, const hipComplex *x,
          int64_t incx, hipComplex *c, int64_t ldc) {
  // Start
  hipblasCdgmm_64(handle /*cublasHandle_t*/, left_right /*cublasSideMode_t*/,
                 m /*int64_t*/, n /*int64_t*/, a /*const cuComplex **/,
                 lda /*int64_t*/, x /*const cuComplex **/, incx /*int64_t*/,
                 c /*cuComplex **/, ldc /*int64_t*/);
  // End
}
